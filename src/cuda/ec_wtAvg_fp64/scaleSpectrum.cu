#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// scaleSpectrum.cu
//
// Code generation for function 'scaleSpectrum'
//

// Include files
#include "scaleSpectrum.h"
#include "div.h"
#include "ec_wtAvg_fp_data.h"
#include "ec_wtAvg_fp_emxutil.h"
#include "ec_wtAvg_fp_mexutil.h"
#include "ec_wtAvg_fp_types.h"
#include "numCpsi.h"
#include "rt_nonfinite.h"
#include "vvarstd.h"
#include "wt.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cmath>
#include <cstdlib>

// Variable Definitions
static emlrtRTEInfo cd_emlrtRTEI{
    12,                                                               // lineNo
    5,                                                                // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo dd_emlrtRTEI{
    211,                                                              // lineNo
    24,                                                               // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo ed_emlrtRTEI{
    96,                                                               // lineNo
    5,                                                                // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo gd_emlrtRTEI{
    298,             // lineNo
    18,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo hd_emlrtRTEI{
    297,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo id_emlrtRTEI{
    12,            // lineNo
    12,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo jd_emlrtRTEI{
    22,            // lineNo
    32,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo kd_emlrtRTEI{
    12,            // lineNo
    1,             // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo ld_emlrtRTEI{
    26,            // lineNo
    43,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo md_emlrtRTEI{
    26,            // lineNo
    37,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo nd_emlrtRTEI{
    293,             // lineNo
    14,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

// Function Declarations
static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line);

static void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                           const char_T *errorName, const char_T *errorString);

static real_T rt_hypotd_snf(real_T u0, real_T u1);

// Function Definitions
static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line)
{
  const char *errName;
  const char *errString;
  nvtxRangePushA("#fcn#cublasCheck#" MW_AT_LOCATION);
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    nvtxMarkA("#raiseCudaError#" MW_AT_LINE);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
  nvtxRangePop();
}

static void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                           const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  nvtxRangePushA("#fcn#raiseCudaError#" MW_AT_LOCATION);
  len = strlen(file);
  pn = static_cast<char_T *>(std::calloc(static_cast<uint32_T>(len + 1UL), 1U));
  fn = static_cast<char_T *>(std::calloc(static_cast<uint32_T>(len + 1UL), 1U));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

static real_T rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T a;
  real_T b;
  real_T y;
  nvtxRangePushA("#fcn#rt_hypotd_snf#" MW_AT_LOCATION);
  a = std::abs(u0);
  b = std::abs(u1);
  if (a < b) {
    a /= b;
    y = b * std::sqrt(a * a + 1.0);
  } else if (a > b) {
    b /= a;
    y = a * std::sqrt(b * b + 1.0);
  } else if (std::isnan(b)) {
    y = rtNaN;
  } else {
    y = a * 1.4142135623730951;
  }
  nvtxRangePop();
  return y;
}

//
//
namespace coder {
void cwtfilterbank_scaleSpectrum(cwtfilterbank *self, const emxArray_real_T *x,
                                 emxArray_real_T *cpu_savgp,
                                 boolean_T *savgp_outdatedOnCpu,
                                 emxArray_real_T *gpu_savgp,
                                 boolean_T *savgp_outdatedOnGpu)
{
  emxArray_creal_T *cfs;
  emxArray_real_T b_gpu_c;
  emxArray_real_T c_gpu_c;
  emxArray_real_T gpu_abscfssq;
  emxArray_real_T gpu_c;
  emxArray_real_T gpu_y;
  emxArray_real_T gpu_z;
  emxArray_real_T *a;
  emxArray_real_T *abswt2S;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *c_cpu_c;
  emxArray_real_T *c_y;
  emxArray_real_T *cpu_abscfssq;
  emxArray_real_T *cpu_c;
  emxArray_real_T *cpu_y;
  emxArray_real_T *cpu_z;
  emxArray_real_T *d_y;
  emxArray_real_T *y;
  emxArray_uint32_T *b_y;
  real_T alpha1;
  real_T b_cpu_z;
  real_T c1;
  real_T normfac;
  real_T *b_gpu_z;
  int32_T acoef;
  int32_T nx;
  int32_T u1;
  uint32_T defaultSL_idx_1;
  boolean_T abscfssq_outdatedOnGpu;
  boolean_T z_needsGpuEnsureCapacity;
  boolean_T z_outdatedOnGpu;
  nvtxRangePushA("#fcn#cwtfilterbank_scaleSpectrum#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_c);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&b_gpu_z, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_c);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_z);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_abscfssq);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  defaultSL_idx_1 = static_cast<uint32_T>(self->Scales->size[1]);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cfs, 2, &o_emlrtRTEI, true);
  nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
  cwtfilterbank_wt(self, x, cfs);
  nvtxMarkA("#vvarstd#" MW_AT_LINE);
  normfac = vvarstd(x, x->size[0]);
  nx = cfs->size[0] * cfs->size[1];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&y, 2, &id_emlrtRTEI, true);
  acoef = y->size[0] * y->size[1];
  y->size[0] = cfs->size[0];
  y->size[1] = cfs->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(y, acoef, &f_emlrtRTEI);
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_0", __LINE__, (nx - 1) + 1,
                   "");
  for (int32_T k{0}; k < nx; k++) {
    nvtxMarkA("#rt_hypotd_snf#" MW_AT_LINE);
    y->data[k] = rt_hypotd_snf(cfs->data[k].re, cfs->data[k].im);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&a, 2, &jd_emlrtRTEI, true);
  acoef = a->size[0] * a->size[1];
  a->size[0] = y->size[0];
  a->size[1] = y->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(a, acoef, &h_emlrtRTEI);
  nx = y->size[0] * y->size[1];
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_1", __LINE__, (nx - 1) + 1,
                   "");
  for (int32_T k{0}; k < nx; k++) {
    a->data[k] = y->data[k] * y->data[k];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&y);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_abscfssq, 2, &kd_emlrtRTEI, true);
  abscfssq_outdatedOnGpu = false;
  acoef = cpu_abscfssq->size[0] * cpu_abscfssq->size[1];
  nx = self->Scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (self->Scales->size[1] == 1) {
    cpu_abscfssq->size[0] = a->size[0];
  } else if (a->size[0] == 1) {
    cpu_abscfssq->size[0] = self->Scales->size[1];
  } else if (a->size[0] == self->Scales->size[1]) {
    cpu_abscfssq->size[0] = a->size[0];
  } else {
    cpu_abscfssq->size[0] = u1;
  }
  cpu_abscfssq->size[1] = a->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_abscfssq, acoef, &i_emlrtRTEI);
  nx = self->Scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (self->Scales->size[1] == 1) {
    u1 = a->size[0];
  } else if (a->size[0] == 1) {
    u1 = self->Scales->size[1];
  } else if (a->size[0] == self->Scales->size[1]) {
    u1 = a->size[0];
  }
  if ((u1 != 0) && (a->size[1] != 0)) {
    int32_T bcoef;
    nx = (a->size[1] != 1);
    u1 = a->size[1] - 1;
    acoef = (a->size[0] != 1);
    bcoef = (self->Scales->size[1] != 1);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_2", __LINE__, u1 + 1,
                     "");
    for (int32_T k{0}; k <= u1; k++) {
      int32_T i1;
      int32_T varargin_2;
      varargin_2 = nx * k;
      i1 = cpu_abscfssq->size[0] - 1;
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_3", __LINE__, i1 + 1,
                       "");
      for (int32_T b_k{0}; b_k <= i1; b_k++) {
        cpu_abscfssq->data[b_k + cpu_abscfssq->size[0] * k] =
            a->data[acoef * b_k + a->size[0] * varargin_2] /
            self->Scales->data[bcoef * b_k];
        abscfssq_outdatedOnGpu = true;
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&a);
  nvtxMarkA("#emxInit_uint32_T#" MW_AT_LINE);
  emxInit_uint32_T(&b_y, 2, &ld_emlrtRTEI, true);
  if (cpu_abscfssq->size[1] < 1) {
    b_y->size[0] = 1;
    b_y->size[1] = 0;
  } else {
    uint32_T u;
    u = static_cast<uint32_T>(cpu_abscfssq->size[1]);
    acoef = b_y->size[0] * b_y->size[1];
    b_y->size[0] = 1;
    b_y->size[1] = cpu_abscfssq->size[1];
    nvtxMarkA("#emxEnsureCapacity_uint32_T#" MW_AT_LINE);
    emxEnsureCapacity_uint32_T(b_y, acoef, &w_emlrtRTEI);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_4", __LINE__,
                     (static_cast<int32_T>(u) - 1) + 1, "");
    for (acoef = 0; acoef < static_cast<int32_T>(u); acoef++) {
      b_y->data[acoef] = static_cast<uint32_T>(acoef) + 1U;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_z, 1, &md_emlrtRTEI, true);
  z_outdatedOnGpu = false;
  acoef = cpu_z->size[0];
  cpu_z->size[0] = cpu_abscfssq->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_z, acoef, &cd_emlrtRTEI);
  z_needsGpuEnsureCapacity = true;
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_5", __LINE__,
                   (cpu_abscfssq->size[0] - 1) + 1, "");
  for (acoef = 0; acoef < cpu_abscfssq->size[0]; acoef++) {
    cpu_z->data[acoef] = 0.0;
    z_outdatedOnGpu = true;
  }
  profileLoopEnd();
  if (cpu_abscfssq->size[1] > 1) {
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_c, 1, &ed_emlrtRTEI, true);
    if (b_y->size[1] == 1) {
      nx = cpu_abscfssq->size[1];
      acoef = cpu_c->size[0];
      cpu_c->size[0] = cpu_abscfssq->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, acoef, &ed_emlrtRTEI);
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_6", __LINE__,
                       (nx - 1) + 1, "");
      for (acoef = 0; acoef < nx; acoef++) {
        cpu_c->data[acoef] = b_y->data[0];
      }
      profileLoopEnd();
      c1 = 0.5 * static_cast<real_T>(b_y->data[0]);
      cpu_c->data[0] = c1;
      cpu_c->data[cpu_abscfssq->size[1] - 1] = c1;
    } else {
      acoef = cpu_c->size[0];
      cpu_c->size[0] = cpu_abscfssq->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, acoef, &dd_emlrtRTEI);
      cpu_c->data[0] =
          0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[1]) -
                                    static_cast<int32_T>(b_y->data[0]));
      nx = cpu_abscfssq->size[1];
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_7", __LINE__,
                       (nx - 3) + 1, "");
      for (int32_T k{0}; k <= nx - 3; k++) {
        cpu_c->data[k + 1] =
            0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[k + 2]) -
                                      static_cast<int32_T>(b_y->data[k]));
      }
      profileLoopEnd();
      cpu_c->data[cpu_abscfssq->size[1] - 1] =
          0.5 * static_cast<real_T>(
                    static_cast<int32_T>(b_y->data[cpu_abscfssq->size[1] - 1]) -
                    static_cast<int32_T>(b_y->data[cpu_abscfssq->size[1] - 2]));
    }
    if (cpu_abscfssq->size[0] >= 1) {
      c1 = 0.0;
      alpha1 = 1.0;
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_abscfssq, &gpu_abscfssq,
                                  !abscfssq_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_c, &gpu_c, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
      z_needsGpuEnsureCapacity = false;
      if (abscfssq_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_abscfssq, cpu_abscfssq);
      }
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_c, cpu_c);
      if (z_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_z, cpu_z);
      }
      nvtxMarkA("#cublasCheck#" MW_AT_LINE);
      cublasCheck(hipblasDgemv(getCublasGlobalHandle(), HIPBLAS_OP_N,
                              cpu_abscfssq->size[0], cpu_abscfssq->size[1],
                              (double *)&alpha1,
                              (double *)&gpu_abscfssq.data[0],
                              cpu_abscfssq->size[0], (double *)&gpu_c.data[0],
                              1, (double *)&c1, (double *)&gpu_z.data[0], 1),
                  __FILE__, __LINE__);
      z_outdatedOnGpu = false;
    }
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_c);
  }
  nvtxMarkA("#emxFree_uint32_T#" MW_AT_LINE);
  emxFree_uint32_T(&b_y);
  b_cpu_z = 0.0;
  abscfssq_outdatedOnGpu = false;
  if (cpu_z->size[0] <= 1) {
    if ((self->Scales->size[1] == 1) && (std::isinf(self->Scales->data[0]) ||
                                         std::isnan(self->Scales->data[0]))) {
      b_cpu_z = rtNaN;
    }
  } else {
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_cpu_c, 1, &ed_emlrtRTEI, true);
    if (self->Scales->size[1] == 1) {
      nx = cpu_z->size[0];
      acoef = b_cpu_c->size[0];
      b_cpu_c->size[0] = cpu_z->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_c, acoef, &ed_emlrtRTEI);
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_8", __LINE__,
                       (nx - 1) + 1, "");
      for (acoef = 0; acoef < nx; acoef++) {
        b_cpu_c->data[acoef] = self->Scales->data[0];
      }
      profileLoopEnd();
      c1 = 0.5 * self->Scales->data[0];
      b_cpu_c->data[0] = c1;
      b_cpu_c->data[cpu_z->size[0] - 1] = c1;
    } else {
      acoef = b_cpu_c->size[0];
      b_cpu_c->size[0] = cpu_z->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_c, acoef, &dd_emlrtRTEI);
      b_cpu_c->data[0] = 0.5 * (self->Scales->data[1] - self->Scales->data[0]);
      nx = cpu_z->size[0];
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_9", __LINE__,
                       (nx - 3) + 1, "");
      for (int32_T k{0}; k <= nx - 3; k++) {
        b_cpu_c->data[k + 1] =
            0.5 * (self->Scales->data[k + 2] - self->Scales->data[k]);
      }
      profileLoopEnd();
      b_cpu_c->data[cpu_z->size[0] - 1] =
          0.5 * (self->Scales->data[cpu_z->size[0] - 1] -
                 self->Scales->data[cpu_z->size[0] - 2]);
    }
    c1 = 0.0;
    alpha1 = 1.0;
    if (z_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(b_cpu_c, &c_gpu_c, false);
    if (z_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_z, cpu_z);
    }
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(&c_gpu_c, b_cpu_c);
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(hipMemcpy(b_gpu_z, &b_cpu_z, 8UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
    nvtxMarkA("#cublasCheck#" MW_AT_LINE);
    cublasCheck(hipblasDgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                            cpu_z->size[0], 1, (double *)&alpha1,
                            (double *)&gpu_z.data[0], cpu_z->size[0],
                            (double *)&c_gpu_c.data[0], 1, (double *)&c1,
                            (double *)b_gpu_z, 1),
                __FILE__, __LINE__);
    abscfssq_outdatedOnGpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_cpu_c);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_z);
  if (!(normfac == 0.0)) {
    if (abscfssq_outdatedOnGpu) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(hipMemcpy(&b_cpu_z, b_gpu_z, 8UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    nvtxMarkA("#numCpsi#" MW_AT_LINE);
    c1 = std::sqrt(normfac /
                   (2.0 /
                    wavelet::internal::cwt::numCpsi(self->Wavelet, self->Gamma,
                                                    self->Beta) /
                    static_cast<real_T>(cpu_abscfssq->size[1]) * b_cpu_z));
    acoef = cfs->size[0] * cfs->size[1];
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_10", __LINE__,
                     (acoef - 1) + 1, "");
    for (u1 = 0; u1 < acoef; u1++) {
      cfs->data[u1].re *= c1;
      cfs->data[u1].im *= c1;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_abscfssq);
  nx = cfs->size[0] * cfs->size[1];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_y, 2, &nd_emlrtRTEI, true);
  acoef = c_y->size[0] * c_y->size[1];
  c_y->size[0] = cfs->size[0];
  c_y->size[1] = cfs->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(c_y, acoef, &f_emlrtRTEI);
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_11", __LINE__,
                   (nx - 1) + 1, "");
  for (int32_T k{0}; k < nx; k++) {
    nvtxMarkA("#rt_hypotd_snf#" MW_AT_LINE);
    c_y->data[k] = rt_hypotd_snf(cfs->data[k].re, cfs->data[k].im);
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cfs);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_y, 2, &nd_emlrtRTEI, true);
  acoef = d_y->size[0] * d_y->size[1];
  d_y->size[0] = c_y->size[0];
  d_y->size[1] = c_y->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(d_y, acoef, &h_emlrtRTEI);
  nx = c_y->size[0] * c_y->size[1];
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_12", __LINE__,
                   (nx - 1) + 1, "");
  for (int32_T k{0}; k < nx; k++) {
    d_y->data[k] = c_y->data[k] * c_y->data[k];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_y);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&abswt2S, 2, &fd_emlrtRTEI, true);
  if (d_y->size[0] == self->Scales->size[1]) {
    acoef = abswt2S->size[0] * abswt2S->size[1];
    abswt2S->size[0] = d_y->size[0];
    abswt2S->size[1] = d_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(abswt2S, acoef, &fd_emlrtRTEI);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_13", __LINE__,
                     (d_y->size[1] - 1) + 1, "");
    for (acoef = 0; acoef < d_y->size[1]; acoef++) {
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_14", __LINE__,
                       (d_y->size[0] - 1) + 1, "");
      for (u1 = 0; u1 < d_y->size[0]; u1++) {
        abswt2S->data[u1 + abswt2S->size[0] * acoef] =
            d_y->data[u1 + d_y->size[0] * acoef] / self->Scales->data[u1];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  } else {
    nvtxMarkA("#binary_expand_op_6#" MW_AT_LINE);
    binary_expand_op_6(abswt2S, d_y, self);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_y);
  nx = static_cast<int32_T>(defaultSL_idx_1);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_y, 2, &gd_emlrtRTEI, true);
  abscfssq_outdatedOnGpu = false;
  acoef = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = static_cast<int32_T>(defaultSL_idx_1);
  cpu_y->size[1] = abswt2S->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_y, acoef, &gd_emlrtRTEI);
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_15", __LINE__,
                   (abswt2S->size[1] - 1) + 1, "");
  for (acoef = 0; acoef < abswt2S->size[1]; acoef++) {
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_16", __LINE__,
                     (nx - 1) + 1, "");
    for (u1 = 0; u1 < nx; u1++) {
      cpu_y->data[u1 + cpu_y->size[0] * acoef] =
          abswt2S->data[u1 + abswt2S->size[0] * acoef];
      abscfssq_outdatedOnGpu = true;
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_c, 1, &ed_emlrtRTEI, true);
  if ((static_cast<int32_T>(defaultSL_idx_1) == 0) && (abswt2S->size[1] == 0)) {
    if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
      b_cpu_z = self->Scales->data[0] * 0.0;
    } else {
      b_cpu_z = 0.0;
    }
    acoef = cpu_savgp->size[0] * cpu_savgp->size[1];
    cpu_savgp->size[0] = 1;
    cpu_savgp->size[1] = 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_savgp, acoef, &hd_emlrtRTEI);
    cpu_savgp->data[0] = b_cpu_z;
    *savgp_outdatedOnCpu = false;
    *savgp_outdatedOnGpu = true;
  } else {
    *savgp_outdatedOnCpu = false;
    *savgp_outdatedOnGpu = false;
    acoef = cpu_savgp->size[0] * cpu_savgp->size[1];
    cpu_savgp->size[0] = 1;
    cpu_savgp->size[1] = abswt2S->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_savgp, acoef, &hd_emlrtRTEI);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_17", __LINE__,
                     (abswt2S->size[1] - 1) + 1, "");
    for (acoef = 0; acoef < abswt2S->size[1]; acoef++) {
      cpu_savgp->data[acoef] = 0.0;
      *savgp_outdatedOnGpu = true;
    }
    profileLoopEnd();
    if (static_cast<int32_T>(defaultSL_idx_1) <= 1) {
      if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
        c1 = self->Scales->data[0];
        if (std::isinf(c1) || std::isnan(c1)) {
          *savgp_outdatedOnGpu = false;
          acoef = cpu_savgp->size[0] * cpu_savgp->size[1];
          cpu_savgp->size[0] = 1;
          cpu_savgp->size[1] = abswt2S->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_savgp, acoef, &hd_emlrtRTEI);
          profileLoopStart("cwtfilterbank_scaleSpectrum_loop_20", __LINE__,
                           (abswt2S->size[1] - 1) + 1, "");
          for (acoef = 0; acoef < abswt2S->size[1]; acoef++) {
            cpu_savgp->data[acoef] = rtNaN;
            *savgp_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
      }
    } else {
      if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
        c1 = self->Scales->data[0];
        acoef = c_cpu_c->size[0];
        c_cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_cpu_c, acoef, &ed_emlrtRTEI);
        profileLoopStart("cwtfilterbank_scaleSpectrum_loop_19", __LINE__,
                         (nx - 1) + 1, "");
        for (acoef = 0; acoef < nx; acoef++) {
          c_cpu_c->data[acoef] = c1;
        }
        profileLoopEnd();
        c1 = 0.5 * self->Scales->data[0];
        c_cpu_c->data[0] = c1;
        c_cpu_c->data[static_cast<int32_T>(defaultSL_idx_1) - 1] = c1;
      } else {
        acoef = c_cpu_c->size[0];
        c_cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_cpu_c, acoef, &dd_emlrtRTEI);
        c_cpu_c->data[0] =
            0.5 * (self->Scales->data[1] - self->Scales->data[0]);
        profileLoopStart("cwtfilterbank_scaleSpectrum_loop_18", __LINE__,
                         (nx - 3) + 1, "");
        for (int32_T k{0}; k <= nx - 3; k++) {
          c_cpu_c->data[k + 1] =
              0.5 * (self->Scales->data[k + 2] - self->Scales->data[k]);
        }
        profileLoopEnd();
        c_cpu_c->data[static_cast<int32_T>(defaultSL_idx_1) - 1] =
            0.5 *
            (self->Scales->data[static_cast<int32_T>(defaultSL_idx_1) - 1] -
             self->Scales->data[static_cast<int32_T>(defaultSL_idx_1) - 2]);
      }
      if (abswt2S->size[1] >= 1) {
        c1 = 0.0;
        alpha1 = 1.0;
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, !abscfssq_outdatedOnGpu);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(c_cpu_c, &b_gpu_c, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_savgp, gpu_savgp,
                                    !*savgp_outdatedOnGpu);
        if (abscfssq_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_y, cpu_y);
        }
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_c, c_cpu_c);
        if (*savgp_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_savgp, cpu_savgp);
        }
        nvtxMarkA("#cublasCheck#" MW_AT_LINE);
        cublasCheck(hipblasDgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                                static_cast<int32_T>(defaultSL_idx_1),
                                abswt2S->size[1], (double *)&alpha1,
                                (double *)&gpu_y.data[0],
                                static_cast<int32_T>(defaultSL_idx_1),
                                (double *)&b_gpu_c.data[0], 1, (double *)&c1,
                                (double *)&gpu_savgp->data[0], 1),
                    __FILE__, __LINE__);
        *savgp_outdatedOnGpu = false;
        *savgp_outdatedOnCpu = true;
      }
    }
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_c);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&abswt2S);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_abscfssq);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_z);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_c);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(b_gpu_z), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_c);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_c);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (scaleSpectrum.cu)

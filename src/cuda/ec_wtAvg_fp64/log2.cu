#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// log2.cu
//
// Code generation for function 'log2'
//

// Include files
#include "log2.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
real_T b_log2(real_T x)
{
  real_T f;
  int32_T eint;
  nvtxRangePushA("#fcn#b_log2#" MW_AT_LOCATION);
  if (x == 0.0) {
    f = rtMinusInf;
  } else if (x < 0.0) {
    f = rtNaN;
  } else if ((!std::isinf(x)) && (!std::isnan(x))) {
    real_T t;
    t = std::frexp(x, &eint);
    if (t == 0.5) {
      f = static_cast<real_T>(eint) - 1.0;
    } else if ((eint == 1) && (t < 0.75)) {
      f = std::log(2.0 * t) / 0.69314718055994529;
    } else {
      f = std::log(t) / 0.69314718055994529 + static_cast<real_T>(eint);
    }
  } else {
    f = x;
  }
  nvtxRangePop();
  return f;
}

} // namespace coder

// End of code generation (log2.cu)

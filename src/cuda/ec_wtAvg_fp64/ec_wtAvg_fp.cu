#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wtAvg_fp.cu
//
// Code generation for function 'ec_wtAvg_fp'
//

// Include files
#include "ec_wtAvg_fp.h"
#include "cwtfilterbank.h"
#include "ec_wtAvg_fp_data.h"
#include "ec_wtAvg_fp_emxutil.h"
#include "ec_wtAvg_fp_mexutil.h"
#include "ec_wtAvg_fp_types.h"
#include "rt_nonfinite.h"
#include "scaleSpectrum.h"
#include "wt.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtMCInfo c_emlrtMCI{
    53,        // lineNo
    19,        // colNo
    "flt2str", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/coder/coder/lib/+coder/+internal/"
    "flt2str.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    29,                                                        // lineNo
    1,                                                         // colNo
    "ec_wtAvg_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtAvg_fp.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    37,                                                        // lineNo
    20,                                                        // colNo
    "ec_wtAvg_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtAvg_fp.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    42,                                                        // lineNo
    26,                                                        // colNo
    "ec_wtAvg_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtAvg_fp.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    42,                                                        // lineNo
    15,                                                        // colNo
    "ec_wtAvg_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtAvg_fp.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    230,             // lineNo
    1,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    75,                                                           // lineNo
    13,                                                           // colNo
    "downsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/downsample.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    287,             // lineNo
    33,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    181,                     // lineNo
    24,                      // colNo
    "combineVectorElements", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
    "combineVectorElements.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    287,             // lineNo
    25,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    27,                                                        // lineNo
    1,                                                         // colNo
    "ec_wtAvg_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtAvg_fp.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    42,                                                        // lineNo
    5,                                                         // colNo
    "ec_wtAvg_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtAvg_fp.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    267,             // lineNo
    36,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    282,             // lineNo
    9,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    287,             // lineNo
    29,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    55,                                                        // lineNo
    10,                                                        // colNo
    "ec_wtAvg_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtAvg_fp.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23]);

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location);

static uint64_T computeNumIters(int32_T ub, int32_T b_ub);

static __global__ void ec_wtAvg_fp_kernel1(const int32_T ch,
                                           const emxArray_real_T x,
                                           const int32_T b_x,
                                           emxArray_real_T xc, int32_T x_dim0);

static __global__ void ec_wtAvg_fp_kernel10(const emxArray_real_T b_dv,
                                            const real_T ds, const int32_T b,
                                            emxArray_real_T c_dv);

static __global__ void ec_wtAvg_fp_kernel11(const emxArray_real_T b_dv,
                                            const int32_T c_dv,
                                            emxArray_real_T d_dv);

static __global__ void ec_wtAvg_fp_kernel12(const emxArray_real_T b_dv,
                                            const int32_T ch, const int32_T b,
                                            emxArray_real_T y, int32_T y_dim0);

static __global__ void
ec_wtAvg_fp_kernel13(const int32_T vlen, const emxArray_real_T x,
                     const int32_T b, emxArray_real_T b_b, int32_T x_dim0);

static __global__ void ec_wtAvg_fp_kernel14(const emxArray_real_T r,
                                            const int32_T b_r,
                                            emxArray_real_T b_dv);

static __global__ void ec_wtAvg_fp_kernel2(const emxArray_creal_T cfs,
                                           const int32_T b, emxArray_real_T y);

static __global__ void ec_wtAvg_fp_kernel3(const emxArray_real_T y,
                                           const int32_T b, emxArray_real_T x);

static __global__ void
ec_wtAvg_fp_kernel4(const real_T nFrames, const emxArray_creal_T cfs,
                    const int32_T nx, const int32_T cfsnorm, const int32_T vlen,
                    emxArray_creal_T b_cfsnorm, int32_T cfs_dim0,
                    int32_T cfsnorm_dim0);

static __global__ void ec_wtAvg_fp_kernel5(const emxArray_creal_T cfsnorm,
                                           const int32_T b,
                                           const int32_T b_cfsnorm,
                                           emxArray_creal_T x, int32_T x_dim0,
                                           int32_T cfsnorm_dim0);

static __global__ void ec_wtAvg_fp_kernel6(const emxArray_creal_T x,
                                           const int32_T b, emxArray_real_T y);

static __global__ void ec_wtAvg_fp_kernel7(const emxArray_real_T y,
                                           const int32_T b, emxArray_real_T x);

static __global__ void ec_wtAvg_fp_kernel8(const int32_T sz, emxArray_real_T b);

static __global__ void ec_wtAvg_fp_kernel9(const emxArray_real_T b,
                                           const real_T nFrames,
                                           const int32_T b_b,
                                           emxArray_real_T b_dv);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23]);

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23]);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23])
{
  static const int32_T dims[2]{1, 23};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 23);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  const mxArray *m3;
  nvtxRangePushA("#fcn#b_sprintf#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  m3 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                             "sprintf", true, location);
  nvtxRangePop();
  return m3;
}

static uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
  }
  numIters *= n;
  nvtxRangePop();
  return numIters;
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel1(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T xc, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    xc.data[xpageoffset] = x.data[xpageoffset + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel10(
    const emxArray_real_T b_dv, const real_T ds, const int32_T b,
    emxArray_real_T c_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    c_dv.data[xpageoffset] = b_dv.data[static_cast<int32_T>(ds) * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel11(
    const emxArray_real_T b_dv, const int32_T c_dv, emxArray_real_T d_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    d_dv.data[xpageoffset] = b_dv.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel12(
    const emxArray_real_T b_dv, const int32_T ch, const int32_T b,
    emxArray_real_T y, int32_T y_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    y.data[xpageoffset + y_dim0 * ch] = b_dv.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel13(
    const int32_T vlen, const emxArray_real_T x, const int32_T b,
    emxArray_real_T b_b, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx);
    xpageoffset = bcoef * x_dim0;
    b_b.data[bcoef] = x.data[xpageoffset];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      b_b.data[bcoef] += x.data[(xpageoffset + k) + 1];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel14(
    const emxArray_real_T r, const int32_T b_r, emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_dv.data[xpageoffset] = r.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel2(
    const emxArray_creal_T cfs, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(cfs.data[k].re, cfs.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel3(
    const emxArray_real_T y, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x.data[k] = y.data[k] * y.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel4(
    const real_T nFrames, const emxArray_creal_T cfs, const int32_T nx,
    const int32_T cfsnorm, const int32_T vlen, emxArray_creal_T b_cfsnorm,
    int32_T cfs_dim0, int32_T cfsnorm_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(vlen) + 1UL) *
                (static_cast<uint64_T>(cfsnorm) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    int32_T bcoef;
    int32_T k;
    int32_T xpageoffset;
    b_k = static_cast<int32_T>(idx % (static_cast<uint64_T>(cfsnorm) + 1UL));
    k = static_cast<int32_T>((idx - static_cast<uint64_T>(b_k)) /
                             (static_cast<uint64_T>(cfsnorm) + 1UL));
    xpageoffset = nx * k + 1;
    bcoef = static_cast<int32_T>(cfs_dim0 != 1);
    b_cfsnorm.data[b_k + cfsnorm_dim0 * k].re =
        nFrames * cfs.data[bcoef * b_k + cfs_dim0 * (xpageoffset - 1)].re;
    b_cfsnorm.data[b_k + cfsnorm_dim0 * k].im =
        nFrames * cfs.data[bcoef * b_k + cfs_dim0 * (xpageoffset - 1)].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel5(
    const emxArray_creal_T cfsnorm, const int32_T b, const int32_T b_cfsnorm,
    emxArray_creal_T x, int32_T x_dim0, int32_T cfsnorm_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_cfsnorm) + 1UL) *
                (static_cast<uint64_T>(b) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b) + 1UL));
    x.data[bcoef + x_dim0 * xpageoffset] =
        cfsnorm.data[bcoef + cfsnorm_dim0 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel6(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel7(
    const emxArray_real_T y, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x.data[k] = y.data[k] * y.data[k];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel8(const int32_T sz,
                                                        emxArray_real_T b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtAvg_fp_kernel9(
    const emxArray_real_T b, const real_T nFrames, const int32_T b_b,
    emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_dv.data[xpageoffset] = nFrames * b.data[xpageoffset];
  }
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23])
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23])
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
  nvtxRangePop();
}

//
// function [y,freqs] = ec_wtAvg_fp(x,fs,fLims,fVoices,ds,doPwr)
void ec_wtAvg_fp(const emxArray_real_T *cpu_x, real_T fs, const real_T fLims[2],
                 real_T fVoices, real_T ds, boolean_T doPwr,
                 emxArray_real_T *cpu_y, emxArray_real_T *freqs)
{
  static const int32_T iv[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  coder::cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal_T c_gpu_x;
  emxArray_creal_T gpu_cfs;
  emxArray_creal_T gpu_cfsnorm;
  emxArray_creal_T *c_cpu_x;
  emxArray_creal_T *cpu_cfs;
  emxArray_creal_T *cpu_cfsnorm;
  emxArray_real_T b_gpu_dv;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_x;
  emxArray_real_T gpu_b;
  emxArray_real_T gpu_dv;
  emxArray_real_T gpu_r;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xc;
  emxArray_real_T gpu_y;
  emxArray_real_T *Scales;
  emxArray_real_T *b_cpu_dv;
  emxArray_real_T *b_cpu_x;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_b;
  emxArray_real_T *cpu_dv;
  emxArray_real_T *cpu_r;
  emxArray_real_T *cpu_xc;
  emxArray_real_T *d_cpu_x;
  const mxArray *b_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *y;
  real_T nFrames;
  int32_T i;
  int32_T xpageoffset;
  char_T unusedExpr[23];
  boolean_T r_outdatedOnCpu;
  boolean_T r_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_wtAvg_fp#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_dv);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_b);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfsnorm);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfs);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xc);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x);
  r_outdatedOnGpu = false;
  x_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInitStruct_cwtfilterbank#" MW_AT_LINE);
  emxInitStruct_cwtfilterbank(&fb, &m_emlrtRTEI, true);
  //  electroCUDA - Scale-avergaged Wavelet Transform
  //    Intended to be compiled into a CUDA mex binary
  //    Kevin Tan, 2024 (github.com/kevmtan/electroCUDA)
  //
  //  OUTPUTS:
  //    y = transformed data
  //    freqs = CWT frequencies
  // 'ec_wtAvg_fp:10' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_wtAvg_fp:11' fs (1,1) double
  //  Sampling rate
  // 'ec_wtAvg_fp:12' fLims (1,2) double
  //  Frequency limits
  // 'ec_wtAvg_fp:13' fVoices (1,1) double = 10
  //  Voices per octave
  // 'ec_wtAvg_fp:14' ds (1,1) double = 0
  //  Downsampling factor
  // 'ec_wtAvg_fp:15' doPwr (1,1) logical = false
  //  Output [0=magnitude|1=output]
  // 'ec_wtAvg_fp:17' if ds<=1
  if (ds <= 1.0) {
    // 'ec_wtAvg_fp:17' ;
    // 'ec_wtAvg_fp:17' ds=0;
    ds = 0.0;
  }
  //  Prep
  // 'ec_wtAvg_fp:20' coder.gpu.kernelfun;
  //  Trigger CUDA kernel creation
  //  Sizes
  // 'ec_wtAvg_fp:23' nFrames = height(x);
  nFrames = cpu_x->size[0];
  // 'ec_wtAvg_fp:24' nChs = width(x);
  //  Make CWT filter
  // 'ec_wtAvg_fp:27' fb =
  // cwtfilterbank(Wavelet="Morse",SignalLength=nFrames,... 'ec_wtAvg_fp:28'
  // SamplingFrequency=fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
  nvtxMarkA("#cwtfilterbank_cwtfilterbank#" MW_AT_LINE);
  coder::cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(cpu_x->size[0]),
                                     fs, fLims, fVoices);
  // 'ec_wtAvg_fp:29' freqs = centerFrequencies(fb);
  xpageoffset = freqs->size[0];
  freqs->size[0] = fb.WaveletCenterFrequencies->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(freqs, xpageoffset, &emlrtRTEI);
  profileLoopStart("ec_wtAvg_fp_loop_0", __LINE__,
                   (fb.WaveletCenterFrequencies->size[0] - 1) + 1, "");
  for (xpageoffset = 0; xpageoffset < fb.WaveletCenterFrequencies->size[0];
       xpageoffset++) {
    freqs->data[xpageoffset] = fb.WaveletCenterFrequencies->data[xpageoffset];
  }
  profileLoopEnd();
  //  Find output frames
  // 'ec_wtAvg_fp:32' if ds
  if (ds != 0.0) {
    // 'ec_wtAvg_fp:33' nFrames = floor(nFrames/ds);
    nFrames = std::floor(static_cast<real_T>(cpu_x->size[0]) / ds);
  }
  //  Preallocate output
  // 'ec_wtAvg_fp:37' y = coder.nullcopy(nan(nFrames,nChs,like=x));
  r_outdatedOnCpu = false;
  xpageoffset = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = static_cast<int32_T>(nFrames);
  cpu_y->size[1] = cpu_x->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_y, xpageoffset, &b_emlrtRTEI);
  y_needsGpuEnsureCapacity = true;
  //  Processing loop across channels
  // 'ec_wtAvg_fp:41' for ch = 1:nChs
  i = cpu_x->size[1];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xc, 1, &c_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv, 1, &n_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&Scales, 2, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfs, 2, &o_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_y, 2, &p_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_x, 2, &p_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfsnorm, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&c_cpu_x, 2, &j_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_y, 2, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_x, 2, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_b, 2, &l_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_r, 2, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_dv, 1, &g_emlrtRTEI, true);
  profileLoopStart("ec_wtAvg_fp_loop_1", __LINE__, (i - 1) + 1, "");
  for (int32_T ch{0}; ch < i; ch++) {
    int32_T nx;
    boolean_T validLaunchParams;
    // 'ec_wtAvg_fp:42' y(:,ch) = awt_lfn(fb,x(:,ch),ds,doPwr);
    xpageoffset = cpu_xc->size[0];
    cpu_xc->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_xc, xpageoffset, &c_emlrtRTEI);
    xpageoffset = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                            2147483647U);
    if (x_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_xc, &gpu_xc, true);
    if (x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_wtAvg_fp_kernel1#" MW_AT_LINE);
      ec_wtAvg_fp_kernel1<<<grid, block>>>(ch, gpu_x, xpageoffset, gpu_xc,
                                           cpu_x->size[0U]);
    }
    //  Run transform %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //  Run transform
    // 'ec_wtAvg_fp:52' if doPwr
    if (doPwr) {
      real_T numfac;
      int32_T vlen;
      uint32_T sz[2];
      uint32_T defaultSL_idx_1;
      // 'ec_wtAvg_fp:53' yc = scaleSpectrum(fb,xc)';
      defaultSL_idx_1 = static_cast<uint32_T>(fb.Scales->size[1]);
      xpageoffset = Scales->size[0] * Scales->size[1];
      Scales->size[0] = 1;
      Scales->size[1] = fb.Scales->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(Scales, xpageoffset, &e_emlrtRTEI);
      profileLoopStart("ec_wtAvg_fp_loop_2", __LINE__,
                       (fb.Scales->size[1] - 1) + 1, "");
      for (xpageoffset = 0; xpageoffset < fb.Scales->size[1]; xpageoffset++) {
        Scales->data[xpageoffset] = fb.Scales->data[xpageoffset];
      }
      profileLoopEnd();
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
      nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
      coder::cwtfilterbank_wt(&fb, cpu_xc, cpu_cfs);
      nx = cpu_cfs->size[0] * cpu_cfs->size[1];
      profileLoopStart("ec_wtAvg_fp_loop_3", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(cpu_cfs->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = static_cast<int32_T>(sz[0]);
      b_cpu_y->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_y, xpageoffset, &f_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfs, &gpu_cfs, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_y, &gpu_y, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_cfs, cpu_cfs);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel2#" MW_AT_LINE);
        ec_wtAvg_fp_kernel2<<<grid, block>>>(gpu_cfs, nx - 1, gpu_y);
      }
      profileLoopStart("ec_wtAvg_fp_loop_4", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(b_cpu_y->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = b_cpu_x->size[0] * b_cpu_x->size[1];
      b_cpu_x->size[0] = static_cast<int32_T>(sz[0]);
      b_cpu_x->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_x, xpageoffset, &h_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(
          computeNumIters(
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
          &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_x, &b_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel3#" MW_AT_LINE);
        ec_wtAvg_fp_kernel3<<<grid, block>>>(
            gpu_y,
            static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
            b_gpu_x);
      }
      vlen = b_cpu_x->size[0] * b_cpu_x->size[1];
      if (b_cpu_x->size[0] * b_cpu_x->size[1] == 0) {
        nFrames = 0.0;
      } else {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(b_cpu_x, &b_gpu_x);
        nFrames = b_cpu_x->data[0];
        profileLoopStart("ec_wtAvg_fp_loop_5", __LINE__, (vlen - 2) + 1, "");
        for (nx = 0; nx <= vlen - 2; nx++) {
          nFrames += b_cpu_x->data[nx + 1];
        }
        profileLoopEnd();
      }
      numfac = fb.sigvar;
      nFrames = std::sqrt(
          numfac / (1.0 / static_cast<real_T>(Scales->size[1]) * nFrames));
      xpageoffset = cpu_cfsnorm->size[0] * cpu_cfsnorm->size[1];
      cpu_cfsnorm->size[0] = cpu_cfs->size[0];
      cpu_cfsnorm->size[1] = cpu_cfs->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsnorm, xpageoffset, &i_emlrtRTEI);
      r_outdatedOnCpu = true;
      if ((cpu_cfs->size[0] != 0) && (cpu_cfs->size[1] != 0)) {
        nx = (cpu_cfs->size[1] != 1);
        vlen = cpu_cfs->size[1] - 1;
        xpageoffset = cpu_cfsnorm->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(vlen, xpageoffset), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_cfsnorm, &gpu_cfsnorm, true);
        r_outdatedOnCpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wtAvg_fp_kernel4#" MW_AT_LINE);
          ec_wtAvg_fp_kernel4<<<grid, block>>>(
              nFrames, gpu_cfs, nx, xpageoffset, vlen, gpu_cfsnorm,
              cpu_cfs->size[0U], cpu_cfsnorm->size[0U]);
        }
      }
      nFrames = 1.0 / static_cast<real_T>(Scales->size[1]);
      xpageoffset = c_cpu_x->size[0] * c_cpu_x->size[1];
      c_cpu_x->size[0] = static_cast<int32_T>(defaultSL_idx_1);
      c_cpu_x->size[1] = cpu_cfsnorm->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(c_cpu_x, xpageoffset, &j_emlrtRTEI);
      xpageoffset = cpu_cfsnorm->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(
          computeNumIters(xpageoffset,
                          static_cast<int32_T>(defaultSL_idx_1) - 1),
          &grid, &block, 2147483647U);
      if (r_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_cfsnorm, &gpu_cfsnorm, true);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(c_cpu_x, &c_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel5#" MW_AT_LINE);
        ec_wtAvg_fp_kernel5<<<grid, block>>>(
            gpu_cfsnorm, static_cast<int32_T>(defaultSL_idx_1) - 1, xpageoffset,
            c_gpu_x, c_cpu_x->size[0U], cpu_cfsnorm->size[0U]);
      }
      nx = static_cast<int32_T>(defaultSL_idx_1) * cpu_cfsnorm->size[1];
      xpageoffset = c_cpu_y->size[0] * c_cpu_y->size[1];
      c_cpu_y->size[0] = static_cast<int32_T>(defaultSL_idx_1);
      c_cpu_y->size[1] = cpu_cfsnorm->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(c_cpu_y, xpageoffset, &f_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(c_cpu_y, &b_gpu_y, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel6#" MW_AT_LINE);
        ec_wtAvg_fp_kernel6<<<grid, block>>>(c_gpu_x, nx - 1, b_gpu_y);
      }
      profileLoopStart("ec_wtAvg_fp_loop_6", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(c_cpu_y->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = d_cpu_x->size[0] * d_cpu_x->size[1];
      d_cpu_x->size[0] = static_cast<int32_T>(sz[0]);
      d_cpu_x->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(d_cpu_x, xpageoffset, &h_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(
          computeNumIters(
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
          &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(d_cpu_x, &d_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel7#" MW_AT_LINE);
        ec_wtAvg_fp_kernel7<<<grid, block>>>(
            b_gpu_y,
            static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
            d_gpu_x);
      }
      vlen = d_cpu_x->size[0];
      if ((d_cpu_x->size[0] == 0) || (d_cpu_x->size[1] == 0)) {
        profileLoopStart("ec_wtAvg_fp_loop_7", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(d_cpu_x->size[xpageoffset]);
        }
        profileLoopEnd();
        xpageoffset = cpu_b->size[0] * cpu_b->size[1];
        cpu_b->size[0] = 1;
        cpu_b->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_b, xpageoffset, &l_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(static_cast<int32_T>(sz[1]) - 1), &grid, &block,
            2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wtAvg_fp_kernel8#" MW_AT_LINE);
          ec_wtAvg_fp_kernel8<<<grid, block>>>(static_cast<int32_T>(sz[1]) - 1,
                                               gpu_b);
        }
      } else {
        nx = d_cpu_x->size[1];
        xpageoffset = cpu_b->size[0] * cpu_b->size[1];
        cpu_b->size[0] = 1;
        cpu_b->size[1] = d_cpu_x->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_b, xpageoffset, &k_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wtAvg_fp_kernel13#" MW_AT_LINE);
          ec_wtAvg_fp_kernel13<<<grid, block>>>(vlen, d_gpu_x, nx - 1, gpu_b,
                                                d_cpu_x->size[0U]);
        }
      }
      xpageoffset = cpu_dv->size[0];
      cpu_dv->size[0] = cpu_b->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_dv, xpageoffset, &d_emlrtRTEI);
      xpageoffset = cpu_b->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel9#" MW_AT_LINE);
        ec_wtAvg_fp_kernel9<<<grid, block>>>(gpu_b, nFrames, xpageoffset,
                                             gpu_dv);
      }
      //  Power
    } else {
      // 'ec_wtAvg_fp:54' else
      // 'ec_wtAvg_fp:55' yc = scaleSpectrum(fb,xc,SpectrumType="density")';
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
      nvtxMarkA("#cwtfilterbank_scaleSpectrum#" MW_AT_LINE);
      coder::cwtfilterbank_scaleSpectrum(&fb, cpu_xc, cpu_r, &r_outdatedOnCpu,
                                         &gpu_r, &r_outdatedOnGpu);
      xpageoffset = cpu_dv->size[0];
      cpu_dv->size[0] = cpu_r->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_dv, xpageoffset, &d_emlrtRTEI);
      xpageoffset = cpu_r->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_r, &gpu_r, !r_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
      if (r_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, cpu_r);
      }
      r_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel14#" MW_AT_LINE);
        ec_wtAvg_fp_kernel14<<<grid, block>>>(gpu_r, xpageoffset, gpu_dv);
      }
      //  Magnitude
    }
    //  Downsample
    // 'ec_wtAvg_fp:59' if ds
    if (ds != 0.0) {
      // 'ec_wtAvg_fp:60' yc = downsample(yc,ds);
      if (!(ds - 1.0 >= 0.0)) {
        y = nullptr;
        m = emlrtCreateCharArray(2, &iv[0]);
        emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
        emlrtAssign(&y, m);
        b_y = nullptr;
        m1 = emlrtCreateDoubleScalar(ds - 1.0);
        emlrtAssign(&b_y, m1);
        nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
        nvtxMarkA("#b_sprintf#" MW_AT_LINE);
        emlrt_marshallIn(b_sprintf(y, b_y, &c_emlrtMCI), "<output of sprintf>",
                         unusedExpr);
      }
      nx = cpu_dv->size[0] - 1;
      xpageoffset = b_cpu_dv->size[0];
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      b_cpu_dv->size[0] =
          div_s32(cpu_dv->size[0] - 1, static_cast<int32_T>(ds)) + 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_dv, xpageoffset, &g_emlrtRTEI);
      xpageoffset = nx / static_cast<int32_T>(ds);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_dv, &b_gpu_dv, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel10#" MW_AT_LINE);
        ec_wtAvg_fp_kernel10<<<grid, block>>>(gpu_dv, ds, xpageoffset,
                                              b_gpu_dv);
      }
      xpageoffset = cpu_dv->size[0];
      cpu_dv->size[0] = b_cpu_dv->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_dv, xpageoffset, &d_emlrtRTEI);
      xpageoffset = b_cpu_dv->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtAvg_fp_kernel11#" MW_AT_LINE);
        ec_wtAvg_fp_kernel11<<<grid, block>>>(b_gpu_dv, xpageoffset, gpu_dv);
      }
    }
    nx = cpu_y->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                            2147483647U);
    if (y_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_y, &c_gpu_y, true);
    }
    y_needsGpuEnsureCapacity = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_wtAvg_fp_kernel12#" MW_AT_LINE);
      ec_wtAvg_fp_kernel12<<<grid, block>>>(gpu_dv, ch, nx - 1, c_gpu_y,
                                            cpu_y->size[0U]);
    }
    r_outdatedOnCpu = true;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_dv);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_r);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_b);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&c_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfsnorm);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfs);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&Scales);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xc);
  nvtxMarkA("#emxFreeStruct_cwtfilterbank#" MW_AT_LINE);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (r_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &c_gpu_y);
  }
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xc);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfs);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfsnorm);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_b);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_dv);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_wtAvg_fp.cu)

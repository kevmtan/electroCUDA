#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// resample.cu
//
// Code generation for function 'resample'
//

// Include files
#include "resample.h"
#include "ResampleParser.h"
#include "conv2.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hip/hip_math_constants.h"
#include <cmath>

// Variable Definitions
static emlrtMCInfo e_emlrtMCI{
    53,        // lineNo
    19,        // colNo
    "flt2str", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/coder/coder/lib/+coder/+internal/"
    "flt2str.m" // pName
};

static emlrtRTEInfo id_emlrtRTEI{
    161,                                                        // lineNo
    71,                                                         // colNo
    "resample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/resample.m" // pName
};

static emlrtRTEInfo jd_emlrtRTEI{
    161,                                                        // lineNo
    9,                                                          // colNo
    "resample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/resample.m" // pName
};

static emlrtRTEInfo kd_emlrtRTEI{
    91,                                                         // lineNo
    5,                                                          // colNo
    "upsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/upsample.m" // pName
};

static emlrtRTEInfo ld_emlrtRTEI{
    1,                // lineNo
    1,                // colNo
    "upfirdnGPUImpl", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+codegenable/+gpu/upfirdnGPUImpl.p" // pName
};

static emlrtRTEInfo od_emlrtRTEI{
    91,                                                           // lineNo
    14,                                                           // colNo
    "downsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/downsample.m" // pName
};

static emlrtRTEInfo pd_emlrtRTEI{
    20,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo qd_emlrtRTEI{
    24,                      // lineNo
    13,                      // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo rd_emlrtRTEI{
    154,                                                        // lineNo
    1,                                                          // colNo
    "resample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/resample.m" // pName
};

static emlrtRTEInfo sd_emlrtRTEI{
    24,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo td_emlrtRTEI{
    88,                                                         // lineNo
    5,                                                          // colNo
    "upsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/upsample.m" // pName
};

static emlrtRTEInfo vd_emlrtRTEI{
    17,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo wd_emlrtRTEI{
    22,                        // lineNo
    23,                        // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23]);

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location);

static
#ifdef __HIPCC__
    __device__
#endif
        int32_T
        div_s32_device(int32_T numerator, int32_T denominator);

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23]);

static __global__ void resample_kernel24(const emxArray_real32_T varargin_1,
                                         const int32_T b_varargin_1,
                                         emxArray_real32_T y);

static __global__ void resample_kernel25(const emxArray_real32_T y,
                                         const int32_T b_y,
                                         emxArray_real32_T varargout_1);

static __global__ void resample_kernel26(const int32_T offsetH, const int32_T b,
                                         int32_T lshift[2]);

static __global__ void resample_kernel27(const int32_T n, const int32_T offsetH,
                                         const int32_T OH, int32_T sizeX[2]);

static __global__ void resample_kernel28(const emxArray_real32_T w,
                                         const int32_T y_dim0, const int32_T n,
                                         const int32_T b, const int32_T c,
                                         emxArray_real32_T b_w, int32_T w_dim0);

static __global__ void resample_kernel29(const emxArray_real32_T w,
                                         const int32_T y,
                                         emxArray_real32_T yOut);

static __global__ void resample_kernel30(emxArray_real_T y);

static __global__ void resample_kernel31(const emxArray_real32_T yOut,
                                         const emxArray_real_T y,
                                         const real_T nlast, const int32_T b,
                                         emxArray_real32_T b_y);

static __global__ void resample_kernel32(const int32_T b, emxArray_real_T y);

static __global__ void resample_kernel33(const int32_T b, emxArray_real32_T w);

static __global__ void resample_kernel34(const real32_T f, emxArray_real32_T w);

static __global__ void resample_kernel35(const int32_T OH,
                                         emxArray_real32_T expanded);

static __global__ void resample_kernel36(const int32_T offsetH,
                                         const emxArray_int32_T y,
                                         const int32_T b_y,
                                         emxArray_int32_T iv2);

static __global__ void
resample_kernel37(const emxArray_real32_T opts_filterWithPadding,
                  const emxArray_int32_T iv2, const int32_T b,
                  emxArray_real32_T expanded);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23])
{
  static const int32_T dims[2]{1, 23};
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 23);
  emlrtDestroyArray(&src);
}

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  pArrays[0] = m1;
  pArrays[1] = m2;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                               "sprintf", true, location);
}

static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if (static_cast<int32_T>(numerator < 0) !=
        static_cast<int32_T>(denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23])
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23])
{
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel24(
    const emxArray_real32_T varargin_1, const int32_T b_varargin_1,
    emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_varargin_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    y.data[orow] = varargin_1.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel25(
    const emxArray_real32_T y, const int32_T b_y, emxArray_real32_T varargout_1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    varargout_1.data[orow] = y.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel26(
    const int32_T offsetH, const int32_T b, int32_T lshift[2])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T h;
    h = lshift[0];
    lshift[0] = lshift[1 - offsetH];
    lshift[1 - offsetH] = h;
  }
}

static __global__ __launch_bounds__(32, 1) void resample_kernel27(
    const int32_T n, const int32_T offsetH, const int32_T OH, int32_T sizeX[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    sizeX[OH - 1] = div_s32_device(offsetH, n) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel28(
    const emxArray_real32_T w, const int32_T y_dim0, const int32_T n,
    const int32_T b, const int32_T c, emxArray_real32_T b_w, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T h;
    int32_T orow;
    h = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    orow = static_cast<int32_T>((idx - static_cast<uint64_T>(h)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_w.data[h + w_dim0 * orow] = w.data[n * h + y_dim0 * orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel29(
    const emxArray_real32_T w, const int32_T y, emxArray_real32_T yOut)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    yOut.data[orow] = w.data[orow];
  }
}

static __global__ __launch_bounds__(32,
                                    1) void resample_kernel30(emxArray_real_T y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    y.data[0] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel31(
    const emxArray_real32_T yOut, const emxArray_real_T y, const real_T nlast,
    const int32_T b, emxArray_real32_T b_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_y.data[orow] = yOut.data[static_cast<int32_T>(nlast + y.data[orow]) - 1];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void resample_kernel32(const int32_T b,
                                                              emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    y.data[orow] = static_cast<real_T>(orow) + 1.0;
  }
}

static __global__
    __launch_bounds__(1024, 1) void resample_kernel33(const int32_T b,
                                                      emxArray_real32_T w)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    w.data[orow] = 0.0F;
  }
}

static __global__
    __launch_bounds__(32, 1) void resample_kernel34(const real32_T f,
                                                    emxArray_real32_T w)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    w.data[0] = f;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel35(
    const int32_T OH, emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel36(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH < 0) && (q1 < MIN_int32_T - offsetH)) {
      q1 = MIN_int32_T;
    } else if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv2.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel37(
    const emxArray_real32_T opts_filterWithPadding, const emxArray_int32_T iv2,
    const int32_T b, emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv2.data[orow]] = opts_filterWithPadding.data[orow];
  }
}

//
//
namespace coder {
void resample(emxArray_real32_T *cpu_varargin_1,
              emxArray_real32_T *gpu_varargin_1,
              boolean_T *varargin_1_outdatedOnGpu, real_T varargin_2,
              real_T varargin_3, emxArray_real32_T *cpu_varargout_1,
              boolean_T *varargout_1_outdatedOnCpu,
              emxArray_real32_T *gpu_varargout_1,
              boolean_T *varargout_1_outdatedOnGpu)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  dim3 block;
  dim3 grid;
  emxArray_int32_T c_gpu_y;
  emxArray_int32_T gpu_iv2;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv2;
  emxArray_int32_T *rows;
  emxArray_real32_T b_gpu_w;
  emxArray_real32_T b_yCol;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T gpu_opts_filterWithPadding;
  emxArray_real32_T gpu_w;
  emxArray_real32_T gpu_y;
  emxArray_real32_T gpu_yOut;
  emxArray_real32_T *b_cpu_w;
  emxArray_real32_T *cpu_expanded;
  emxArray_real32_T *cpu_opts_filterWithPadding;
  emxArray_real32_T *cpu_w;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *cpu_yOut;
  emxArray_real32_T *opts_filter;
  emxArray_real32_T *yCol;
  emxArray_real_T b_gpu_y;
  emxArray_real_T *c_cpu_y;
  const mxArray *b_y;
  const mxArray *e_y;
  const mxArray *f_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T absx;
  real_T dlast;
  real_T n;
  real_T nlast;
  real_T x;
  real_T xin;
  int32_T d_y[2];
  int32_T(*gpu_lshift)[2];
  int32_T(*gpu_sizeX)[2];
  int32_T k;
  int32_T mc;
  char_T unusedExpr[23];
  boolean_T d_opts_filterWithPadding_outdat;
  boolean_T validLaunchParams;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  checkCudaError(mwCudaMalloc(&gpu_lshift, 8UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_sizeX, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real32_T(&b_gpu_w);
  gpuEmxReset_real_T(&b_gpu_y);
  gpuEmxReset_real32_T(&gpu_yOut);
  gpuEmxReset_int32_T(&gpu_iv2);
  gpuEmxReset_int32_T(&c_gpu_y);
  gpuEmxReset_real32_T(&gpu_expanded);
  gpuEmxReset_real32_T(&gpu_w);
  gpuEmxReset_real32_T(&gpu_y);
  gpuEmxReset_real32_T(&gpu_opts_filterWithPadding);
  w_outdatedOnGpu = false;
  y_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real32_T(&opts_filter, 1, &rd_emlrtRTEI, true);
  emxInit_real32_T(&cpu_opts_filterWithPadding, 1, &rd_emlrtRTEI, true);
  opts_filter->size[0] = 0;
  cpu_opts_filterWithPadding->size[0] = 0;
  x = varargin_2 / varargin_3;
  xin = x;
  absx = std::abs(x);
  if (std::isinf(absx) || std::isnan(absx)) {
    absx = rtNaN;
  } else if (absx < 4.4501477170144028E-308) {
    absx = 4.94065645841247E-324;
  } else {
    std::frexp(absx, &mc);
    absx = std::ldexp(1.0, mc - 53);
  }
  absx = std::fmax(1.0E-12, absx);
  if (std::isinf(x) || std::isnan(x)) {
    if (!std::isnan(x)) {
      if (x < 0.0) {
        n = -1.0;
      } else {
        n = 1.0;
      }
    } else {
      n = 0.0;
    }
    absx = 0.0;
  } else {
    real_T d;
    n = 1.0;
    d = 0.0;
    nlast = 0.0;
    dlast = 1.0;
    real_T rndx;
    int32_T exitg1;
    do {
      exitg1 = 0;
      rndx = std::round(x);
      if (!std::isinf(x)) {
        real_T tmp;
        x -= rndx;
        tmp = nlast;
        nlast = n;
        n = n * rndx + tmp;
        tmp = dlast;
        dlast = d;
        d = d * rndx + tmp;
      } else {
        nlast = n;
        dlast = d;
        n = x;
        d = 0.0;
      }
      if ((x == 0.0) || (std::abs(n / d - xin) <= absx)) {
        exitg1 = 1;
      } else {
        x = 1.0 / x;
      }
    } while (exitg1 == 0);
    if (std::isnan(d)) {
      absx = rtNaN;
    } else if (d < 0.0) {
      absx = -1.0;
    } else {
      absx = (d > 0.0);
    }
    n /= absx;
    absx = std::abs(d);
  }
  emxInit_real32_T(&cpu_y, 1, &sd_emlrtRTEI, true);
  k = cpu_y->size[0];
  cpu_y->size[0] = cpu_varargin_1->size[0];
  emxEnsureCapacity_real32_T(cpu_y, k, &id_emlrtRTEI);
  k = cpu_varargin_1->size[0] - 1;
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real32_T(cpu_varargin_1, gpu_varargin_1,
                                !*varargin_1_outdatedOnGpu);
  gpuEmxEnsureCapacity_real32_T(cpu_y, &gpu_y, true);
  if (*varargin_1_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_real32_T(gpu_varargin_1, cpu_varargin_1);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel24<<<grid, block>>>(*gpu_varargin_1, k, gpu_y);
  }
  gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &gpu_y);
  b_signal::internal::resample::c_ResampleParser_createFilterAn(
      &n, &absx, opts_filter, cpu_opts_filterWithPadding, &nlast, cpu_y);
  d_opts_filterWithPadding_outdat = true;
  w_outdatedOnCpu = true;
  emxFree_real32_T(&opts_filter);
  emxInit_real32_T(&yCol, 1, &td_emlrtRTEI, true);
  emxInit_real32_T(&cpu_w, 2, &ld_emlrtRTEI, true);
  emxInit_real32_T(&cpu_expanded, 2, &md_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &ud_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv2, 1, &md_emlrtRTEI, true);
  emxInit_real32_T(&cpu_yOut, 2, &vd_emlrtRTEI, true);
  emxInit_int32_T(&rows, 2, &md_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_y, 2, &wd_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_w, 2, &od_emlrtRTEI, true);
  if ((!(n == 1.0)) || (!(absx == 1.0))) {
    int32_T cpu_lshift[2];
    int32_T cpu_sizeX[2];
    int32_T OH;
    int32_T b_n;
    int32_T offsetH;
    boolean_T c_opts_filterWithPadding_outdat;
    boolean_T w_needsGpuEnsureCapacity;
    dlast = std::ceil(static_cast<real_T>(cpu_y->size[0]) * n / absx);
    if (!(n - 1.0 >= 0.0)) {
      y = nullptr;
      m = emlrtCreateCharArray(2, &iv[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
      emlrtAssign(&y, m);
      b_y = nullptr;
      m1 = emlrtCreateDoubleScalar(n - 1.0);
      emlrtAssign(&b_y, m1);
      emlrt_marshallIn(b_sprintf(y, b_y, &e_emlrtMCI), "<output of sprintf>",
                       unusedExpr);
    }
    k = yCol->size[0];
    yCol->size[0] = cpu_y->size[0] * static_cast<int32_T>(n);
    emxEnsureCapacity_real32_T(yCol, k, &kd_emlrtRTEI);
    for (offsetH = 0; offsetH < cpu_y->size[0] * static_cast<int32_T>(n);
         offsetH++) {
      yCol->data[offsetH] = 0.0F;
    }
    if ((static_cast<int32_T>(n) == 0) ||
        (((static_cast<int32_T>(n) > 0) &&
          (cpu_y->size[0] * static_cast<int32_T>(n) < 1)) ||
         ((static_cast<int32_T>(n) < 0) &&
          (cpu_y->size[0] * static_cast<int32_T>(n) > 1)))) {
      b_n = 1;
      mc = 0;
    } else {
      b_n = static_cast<int32_T>(n);
      mc = cpu_y->size[0] * static_cast<int32_T>(n);
    }
    mc = div_s32(mc - 1, b_n) + 1;
    for (offsetH = 0; offsetH < mc; offsetH++) {
      yCol->data[b_n * offsetH] = cpu_y->data[offsetH];
    }
    if ((cpu_y->size[0] * static_cast<int32_T>(n) == 1) &&
        (cpu_opts_filterWithPadding->size[0] == 1)) {
      k = cpu_w->size[0] * cpu_w->size[1];
      cpu_w->size[0] = 1;
      cpu_w->size[1] = 1;
      emxEnsureCapacity_real32_T(cpu_w, k, &ld_emlrtRTEI);
      cpu_w->data[0] = yCol->data[0] * cpu_opts_filterWithPadding->data[0];
      w_outdatedOnGpu = true;
      w_needsGpuEnsureCapacity = true;
    } else if (cpu_opts_filterWithPadding->size[0] >
               cpu_y->size[0] * static_cast<int32_T>(n)) {
      if (cpu_y->size[0] * static_cast<int32_T>(n) == 0) {
        mc = cpu_opts_filterWithPadding->size[0] +
             cpu_y->size[0] * static_cast<int32_T>(n);
      } else {
        mc = (cpu_opts_filterWithPadding->size[0] +
              cpu_y->size[0] * static_cast<int32_T>(n)) -
             1;
      }
      k = cpu_w->size[0] * cpu_w->size[1];
      cpu_w->size[0] = mc;
      cpu_w->size[1] = 1;
      emxEnsureCapacity_real32_T(cpu_w, k, &ld_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_w, &gpu_w, true);
      w_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel33<<<grid, block>>>(mc - 1, gpu_w);
      }
      w_outdatedOnCpu = true;
      if (cpu_y->size[0] * static_cast<int32_T>(n) != 0) {
        if ((cpu_opts_filterWithPadding->size[0] == 1) &&
            (cpu_y->size[0] * static_cast<int32_T>(n) == 1)) {
          real32_T b_cv;
          k = cpu_w->size[0] * cpu_w->size[1];
          cpu_w->size[0] = 1;
          cpu_w->size[1] = 1;
          emxEnsureCapacity_real32_T(cpu_w, k, &ld_emlrtRTEI);
          b_cv = cpu_opts_filterWithPadding->data[0] * yCol->data[0];
          gpuEmxEnsureCapacity_real32_T(cpu_w, &gpu_w, true);
          resample_kernel34<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b_cv,
                                                                     gpu_w);
        } else {
          uint32_T b_OH;
          b_OH = (static_cast<uint32_T>(cpu_opts_filterWithPadding->size[0]) +
                  static_cast<uint32_T>(cpu_y->size[0] *
                                        static_cast<int32_T>(n))) -
                 1U;
          if (b_OH > 2147483647U) {
            b_OH = 2147483647U;
          }
          OH = static_cast<int32_T>(b_OH);
          offsetH =
              static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_y->size[0] *
                                                 static_cast<int32_T>(n)) /
                             2.0)) +
              static_cast<int32_T>(
                  std::floor((static_cast<real_T>(cpu_y->size[0] *
                                                  static_cast<int32_T>(n)) -
                              1.0) /
                             2.0));
          mc = cpu_y->size[0] * static_cast<int32_T>(n);
          if ((static_cast<int32_T>(b_OH) > 0) &&
              (mc > MAX_int32_T - static_cast<int32_T>(b_OH))) {
            mc = MAX_int32_T;
          } else {
            mc += static_cast<int32_T>(b_OH);
          }
          k = cpu_expanded->size[0] * cpu_expanded->size[1];
          cpu_expanded->size[0] = mc - 1;
          cpu_expanded->size[1] = 1;
          emxEnsureCapacity_real32_T(cpu_expanded, k, &md_emlrtRTEI);
          k = (static_cast<int32_T>(b_OH) +
               cpu_y->size[0] * static_cast<int32_T>(n)) -
              2;
          mwGetLaunchParameters1D(computeNumIters(k), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            resample_kernel35<<<grid, block>>>(k, gpu_expanded);
          }
          b_n = cpu_opts_filterWithPadding->size[0];
          k = b_cpu_y->size[0] * b_cpu_y->size[1];
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = cpu_opts_filterWithPadding->size[0];
          emxEnsureCapacity_int32_T(b_cpu_y, k, &nd_emlrtRTEI);
          b_cpu_y->data[0] = 1;
          mc = 1;
          for (k = 0; k <= b_n - 2; k++) {
            mc++;
            b_cpu_y->data[k + 1] = mc;
          }
          k = cpu_iv2->size[0];
          cpu_iv2->size[0] = b_cpu_y->size[1];
          emxEnsureCapacity_int32_T(cpu_iv2, k, &md_emlrtRTEI);
          k = b_cpu_y->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(k), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_int32_T(b_cpu_y, &c_gpu_y, false);
          gpuEmxEnsureCapacity_int32_T(cpu_iv2, &gpu_iv2, true);
          gpuEmxMemcpyCpuToGpu_int32_T(&c_gpu_y, b_cpu_y);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            resample_kernel36<<<grid, block>>>(offsetH, c_gpu_y, k, gpu_iv2);
          }
          mc = cpu_iv2->size[0];
          mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_opts_filterWithPadding,
                                        &gpu_opts_filterWithPadding, false);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_opts_filterWithPadding,
                                        cpu_opts_filterWithPadding);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            resample_kernel37<<<grid, block>>>(gpu_opts_filterWithPadding,
                                               gpu_iv2, mc - 1, gpu_expanded);
          }
          c_opts_filterWithPadding_outdat = true;
          mc = cpu_y->size[0] * static_cast<int32_T>(n);
          if (mc - 1 < 0) {
            b_n = 0;
          } else {
            b_n = mc;
          }
          k = rows->size[0] * rows->size[1];
          rows->size[0] = 1;
          rows->size[1] = b_n;
          emxEnsureCapacity_int32_T(rows, k, &nd_emlrtRTEI);
          if (b_n > 0) {
            rows->data[0] = 0;
            mc = 0;
            for (k = 0; k <= b_n - 2; k++) {
              mc++;
              rows->data[k + 1] = mc;
            }
          }
          k = cpu_w->size[0] * cpu_w->size[1];
          cpu_w->size[0] = static_cast<int32_T>(b_OH);
          cpu_w->size[1] = 1;
          emxEnsureCapacity_real32_T(cpu_w, k, &md_emlrtRTEI);
          w_needsGpuEnsureCapacity = true;
          if (static_cast<int32_T>(b_OH) != 0) {
            for (offsetH = 0; offsetH < OH; offsetH++) {
              real32_T b_cv;
              b_cv = 0.0F;
              k = cpu_y->size[0] * static_cast<int32_T>(n);
              for (b_n = 0; b_n < k; b_n++) {
                mc = rows->data[b_n];
                if (mc > 2147483646 - offsetH) {
                  mc = MAX_int32_T;
                } else {
                  mc = (offsetH + mc) + 1;
                }
                if (c_opts_filterWithPadding_outdat) {
                  gpuEmxMemcpyGpuToCpu_real32_T(cpu_expanded, &gpu_expanded);
                }
                c_opts_filterWithPadding_outdat = false;
                b_cv += cpu_expanded->data[mc - 1] * yCol->data[(k - b_n) - 1];
              }
              if (w_outdatedOnCpu) {
                gpuEmxMemcpyGpuToCpu_real32_T(cpu_w, &gpu_w);
              }
              cpu_w->data[offsetH] = b_cv;
              w_outdatedOnCpu = false;
              w_outdatedOnGpu = true;
            }
          }
        }
      }
    } else {
      int32_T c_y[2];
      c_y[0] = cpu_y->size[0] * static_cast<int32_T>(n);
      b_yCol = *yCol;
      d_y[0] = c_y[0];
      d_y[1] = 1;
      b_yCol.size = &d_y[0];
      b_yCol.numDimensions = 2;
      conv2NonSeparable(&b_yCol, cpu_opts_filterWithPadding,
                        &gpu_opts_filterWithPadding,
                        &d_opts_filterWithPadding_outdat, cpu_w,
                        &w_outdatedOnCpu, &gpu_w, &w_outdatedOnGpu);
      w_needsGpuEnsureCapacity = true;
    }
    if (!(absx - 1.0 >= 0.0)) {
      e_y = nullptr;
      m2 = emlrtCreateCharArray(2, &iv1[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
      emlrtAssign(&e_y, m2);
      f_y = nullptr;
      m3 = emlrtCreateDoubleScalar(absx - 1.0);
      emlrtAssign(&f_y, m3);
      emlrt_marshallIn(b_sprintf(e_y, f_y, &e_emlrtMCI), "<output of sprintf>",
                       unusedExpr);
    }
    OH = 2;
    if (cpu_w->size[0] != 1) {
      OH = 1;
    }
    d_opts_filterWithPadding_outdat = false;
    c_opts_filterWithPadding_outdat = false;
    for (offsetH = 0; offsetH < 2; offsetH++) {
      cpu_sizeX[offsetH] = cpu_w->size[offsetH];
      c_opts_filterWithPadding_outdat = true;
      cpu_lshift[offsetH] = cpu_w->size[offsetH];
      d_opts_filterWithPadding_outdat = true;
    }
    if (1 - OH >= 0) {
      mwGetLaunchParameters1D(
          computeNumIters(static_cast<int32_T>(
                              (2.0 - (1.0 - static_cast<real_T>(OH))) / 2.0) -
                          1),
          &grid, &block, 2147483647U);
      if (d_opts_filterWithPadding_outdat) {
        checkCudaError(
            hipMemcpy(*gpu_lshift, cpu_lshift, 8UL, hipMemcpyHostToDevice),
            __FILE__, __LINE__);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel26<<<grid, block>>>(
            1 - OH,
            static_cast<int32_T>((2.0 - (1.0 - static_cast<real_T>(OH))) /
                                 2.0) -
                1,
            *gpu_lshift);
      }
      checkCudaError(
          hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
          __FILE__, __LINE__);
    }
    mc = cpu_lshift[0];
    cpu_lshift[0] = cpu_lshift[1];
    if ((absx == 0.0) || ((absx > 0.0) && (cpu_lshift[0] < 1))) {
      b_n = 1;
      offsetH = -1;
    } else {
      b_n = static_cast<int32_T>(absx);
      offsetH = cpu_lshift[0] - 1;
    }
    if (c_opts_filterWithPadding_outdat) {
      checkCudaError(
          hipMemcpy(*gpu_sizeX, cpu_sizeX, 8UL, hipMemcpyHostToDevice),
          __FILE__, __LINE__);
    }
    resample_kernel27<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b_n, offsetH, OH,
                                                               *gpu_sizeX);
    k = b_cpu_w->size[0] * b_cpu_w->size[1];
    b_cpu_w->size[0] = div_s32(offsetH, b_n) + 1;
    b_cpu_w->size[1] = mc;
    emxEnsureCapacity_real32_T(b_cpu_w, k, &od_emlrtRTEI);
    k = offsetH / b_n;
    mwGetLaunchParameters1D(computeNumIters(mc - 1, k), &grid, &block,
                            2147483647U);
    if (w_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real32_T(b_cpu_w, &b_gpu_w, true);
    if (w_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_w, cpu_w);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      resample_kernel28<<<grid, block>>>(gpu_w, cpu_lshift[0], b_n, k, mc - 1,
                                         b_gpu_w, b_cpu_w->size[0U]);
    }
    checkCudaError(
        hipMemcpy(cpu_sizeX, *gpu_sizeX, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    k = cpu_yOut->size[0] * cpu_yOut->size[1];
    cpu_yOut->size[0] = cpu_sizeX[0];
    cpu_yOut->size[1] = cpu_sizeX[1];
    emxEnsureCapacity_real32_T(cpu_yOut, k, &pd_emlrtRTEI);
    k = cpu_sizeX[0] * cpu_sizeX[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_yOut, &gpu_yOut, true);
    c_opts_filterWithPadding_outdat = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      resample_kernel29<<<grid, block>>>(b_gpu_w, k, gpu_yOut);
    }
    w_needsGpuEnsureCapacity = false;
    d_opts_filterWithPadding_outdat = true;
    absx =
        std::ceil(((static_cast<real_T>(cpu_y->size[0]) - 1.0) * n +
                   static_cast<real_T>(cpu_opts_filterWithPadding->size[0])) /
                  absx);
    if (!(cpu_yOut->size[0] < absx)) {
      if (absx < 1.0) {
        mc = 0;
      } else {
        mc = static_cast<int32_T>(absx);
      }
      b_n = cpu_yOut->size[1];
      for (k = 0; k < b_n; k++) {
        for (offsetH = 0; offsetH < mc; offsetH++) {
          if (d_opts_filterWithPadding_outdat) {
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_yOut, &gpu_yOut);
          }
          cpu_yOut->data[offsetH + mc * k] =
              cpu_yOut->data[offsetH + cpu_yOut->size[0] * k];
          d_opts_filterWithPadding_outdat = false;
          w_needsGpuEnsureCapacity = true;
        }
      }
      k = cpu_yOut->size[0] * cpu_yOut->size[1];
      cpu_yOut->size[0] = mc;
      emxEnsureCapacity_real32_T(cpu_yOut, k, &pd_emlrtRTEI);
      c_opts_filterWithPadding_outdat = true;
    }
    if (std::isnan(dlast)) {
      k = c_cpu_y->size[0] * c_cpu_y->size[1];
      c_cpu_y->size[0] = 1;
      c_cpu_y->size[1] = 1;
      emxEnsureCapacity_real_T(c_cpu_y, k, &gb_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(c_cpu_y, &b_gpu_y, true);
      y_needsGpuEnsureCapacity = false;
      resample_kernel30<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b_gpu_y);
    } else if (dlast < 1.0) {
      c_cpu_y->size[0] = 1;
      c_cpu_y->size[1] = 0;
    } else {
      k = c_cpu_y->size[0] * c_cpu_y->size[1];
      c_cpu_y->size[0] = 1;
      c_cpu_y->size[1] = static_cast<int32_T>(dlast - 1.0) + 1;
      emxEnsureCapacity_real_T(c_cpu_y, k, &gb_emlrtRTEI);
      mwGetLaunchParameters1D(
          computeNumIters(static_cast<int32_T>(dlast - 1.0)), &grid, &block,
          2147483647U);
      gpuEmxEnsureCapacity_real_T(c_cpu_y, &b_gpu_y, true);
      y_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel32<<<grid, block>>>(static_cast<int32_T>(dlast - 1.0),
                                           b_gpu_y);
      }
    }
    k = cpu_y->size[0];
    cpu_y->size[0] = static_cast<int32_T>(dlast);
    emxEnsureCapacity_real32_T(cpu_y, k, &qd_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(dlast) - 1),
                            &grid, &block, 2147483647U);
    if (c_opts_filterWithPadding_outdat) {
      gpuEmxEnsureCapacity_real32_T(cpu_yOut, &gpu_yOut,
                                    !w_needsGpuEnsureCapacity);
    }
    if (y_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(c_cpu_y, &b_gpu_y, true);
    }
    gpuEmxEnsureCapacity_real32_T(cpu_y, &gpu_y, true);
    w_outdatedOnCpu = false;
    if (w_needsGpuEnsureCapacity) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_yOut, cpu_yOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      resample_kernel31<<<grid, block>>>(
          gpu_yOut, b_gpu_y, nlast, static_cast<int32_T>(dlast) - 1, gpu_y);
    }
  }
  emxFree_real32_T(&b_cpu_w);
  emxFree_real_T(&c_cpu_y);
  emxFree_int32_T(&rows);
  emxFree_real32_T(&cpu_yOut);
  emxFree_int32_T(&cpu_iv2);
  emxFree_int32_T(&b_cpu_y);
  emxFree_real32_T(&cpu_expanded);
  emxFree_real32_T(&cpu_w);
  emxFree_real32_T(&yCol);
  emxFree_real32_T(&cpu_opts_filterWithPadding);
  k = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
  cpu_varargout_1->size[0] = cpu_y->size[0];
  cpu_varargout_1->size[1] = 1;
  emxEnsureCapacity_real32_T(cpu_varargout_1, k, &jd_emlrtRTEI);
  k = cpu_y->size[0] - 1;
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  if (w_outdatedOnCpu) {
    gpuEmxEnsureCapacity_real32_T(cpu_y, &gpu_y, true);
  }
  gpuEmxEnsureCapacity_real32_T(cpu_varargout_1, gpu_varargout_1, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel25<<<grid, block>>>(gpu_y, k, *gpu_varargout_1);
  }
  emxFree_real32_T(&cpu_y);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_opts_filterWithPadding);
  gpuEmxFree_real32_T(&gpu_y);
  gpuEmxFree_real32_T(&gpu_w);
  gpuEmxFree_real32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&c_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv2);
  gpuEmxFree_real32_T(&gpu_yOut);
  gpuEmxFree_real_T(&b_gpu_y);
  gpuEmxFree_real32_T(&b_gpu_w);
  checkCudaError(mwCudaFree(*gpu_sizeX), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_lshift), __FILE__, __LINE__);
  *varargin_1_outdatedOnGpu = false;
  *varargout_1_outdatedOnCpu = true;
  *varargout_1_outdatedOnGpu = false;
}

} // namespace coder

// End of code generation (resample.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// vvarstd.cu
//
// Code generation for function 'vvarstd'
//

// Include files
#include "vvarstd.h"
#include "ec_cwtAvg_types.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
real32_T vvarstd(const emxArray_real32_T *v, int32_T n)
{
  real32_T s;
  if (n == 0) {
    s = rtNaNF;
  } else if (n == 1) {
    if ((!std::isinf(v->data[0])) && (!std::isnan(v->data[0]))) {
      s = 0.0F;
    } else {
      s = rtNaNF;
    }
  } else {
    real32_T xbar;
    if (v->size[0] == 0) {
      xbar = 0.0F;
    } else {
      xbar = v->data[0];
      for (int32_T k{0}; k <= n - 2; k++) {
        xbar += v->data[k + 1];
      }
    }
    xbar /= static_cast<real32_T>(n);
    s = 0.0F;
    for (int32_T k{0}; k < n; k++) {
      real32_T t;
      t = v->data[k] - xbar;
      s += t * t;
    }
    s /= static_cast<real32_T>(n);
  }
  return s;
}

} // namespace coder

// End of code generation (vvarstd.cu)

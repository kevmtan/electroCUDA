#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtfilterbank.cu
//
// Code generation for function 'cwtfilterbank'
//

// Include files
#include "cwtfilterbank.h"
#include "bsxfun.h"
#include "cwtfreqlimits.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "log2.h"
#include "morsebpfilters.h"
#include "rt_nonfinite.h"
#include "wavCFandSD.h"
#include "wavbpfilters.h"
#include <cmath>

// Variable Definitions
static emlrtMCInfo emlrtMCI{
    53,                                                                // lineNo
    14,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtMCInfo b_emlrtMCI{
    55,                                                                // lineNo
    15,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtRTEInfo db_emlrtRTEI{
    999,             // lineNo
    17,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo eb_emlrtRTEI{
    1000,            // lineNo
    17,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo fb_emlrtRTEI{
    1009,            // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo hb_emlrtRTEI{
    1278,            // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo ib_emlrtRTEI{
    217,             // lineNo
    21,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo jb_emlrtRTEI{
    18,             // lineNo
    5,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo kb_emlrtRTEI{
    31,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo lb_emlrtRTEI{
    18,               // lineNo
    5,                // colNo
    "morsebpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morsebpfilters.m" // pName
};

static emlrtRTEInfo mb_emlrtRTEI{
    33,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo nb_emlrtRTEI{
    25,               // lineNo
    5,                // colNo
    "morsebpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morsebpfilters.m" // pName
};

static emlrtRTEInfo qb_emlrtRTEI{
    41,             // lineNo
    1,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo sb_emlrtRTEI{
    33,               // lineNo
    1,                // colNo
    "morsebpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morsebpfilters.m" // pName
};

static emlrtRTEInfo tb_emlrtRTEI{
    956,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo ub_emlrtRTEI{
    227,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo vb_emlrtRTEI{
    228,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo wb_emlrtRTEI{
    1276,            // lineNo
    30,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo xb_emlrtRTEI{
    1276,            // lineNo
    25,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo yb_emlrtRTEI{
    16,             // lineNo
    5,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo ac_emlrtRTEI{
    32,             // lineNo
    18,             // colNo
    "getCWTScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "getCWTScales.m" // pName
};

static emlrtRTEInfo bc_emlrtRTEI{
    16,               // lineNo
    5,                // colNo
    "morsebpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morsebpfilters.m" // pName
};

static emlrtRTEInfo cc_emlrtRTEI{
    26,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo dc_emlrtRTEI{
    32,             // lineNo
    13,             // colNo
    "getCWTScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "getCWTScales.m" // pName
};

static emlrtRTEInfo ec_emlrtRTEI{
    21,               // lineNo
    1,                // colNo
    "morsebpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morsebpfilters.m" // pName
};

static emlrtRTEInfo fc_emlrtRTEI{
    34,             // lineNo
    42,             // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo gc_emlrtRTEI{
    175,             // lineNo
    25,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo hc_emlrtRTEI{
    934,             // lineNo
    26,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo ic_emlrtRTEI{
    816,             // lineNo
    25,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret);

namespace coder {
static cwtfilterbank *cwtfilterbank_setProperties(cwtfilterbank *self,
                                                  real_T varargin_4,
                                                  real_T varargin_6,
                                                  const real_T varargin_8[2],
                                                  real_T varargin_10);

}
static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y);

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location);

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret)
{
  static const int32_T dims[2]{1, -1};
  int32_T iv[2];
  int32_T i;
  boolean_T bv[2]{false, true};
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                            (const void *)&dims[0], &bv[0], &iv[0]);
  i = ret->size[0] * ret->size[1];
  ret->size[0] = iv[0];
  ret->size[1] = iv[1];
  emxEnsureCapacity_char_T(ret, i, static_cast<emlrtRTEInfo *>(nullptr));
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret->data[0], 1, false);
  emlrtDestroyArray(&src);
}

//
//
namespace coder {
static cwtfilterbank *cwtfilterbank_setProperties(cwtfilterbank *self,
                                                  real_T varargin_4,
                                                  real_T varargin_6,
                                                  const real_T varargin_8[2],
                                                  real_T varargin_10)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 2};
  static const int32_T iv2[2]{1, 7};
  static const int32_T iv3[2]{1, 5};
  static const char_T b[10]{'r', 'e', 'f', 'l', 'e', 'c', 't', 'i', 'o', 'n'};
  static const char_T cv2[10]{'r', 'e', 'f', 'l', 'e', 'c', 't', 'i', 'o', 'n'};
  static const char_T u[7]{'s', 'p', 'r', 'i', 'n', 't', 'f'};
  static const char_T b_cv[5]{'M', 'o', 'r', 's', 'e'};
  static const char_T b_formatSpec[5]{'%', '2', '.', '2', 'f'};
  static const char_T cv3[5]{'M', 'o', 'r', 's', 'e'};
  static const char_T cv4[5]{'m', 'o', 'r', 's', 'e'};
  static const char_T formatSpec[2]{'%', 'f'};
  cwtfilterbank *b_self;
  emxArray_char_T *d_tmpStr;
  emxArray_char_T *tmpStr;
  const mxArray *b_tmpStr;
  const mxArray *b_y;
  const mxArray *c_tmpStr;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *f_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *m4;
  const mxArray *m5;
  const mxArray *y;
  real_T NyquistRange_idx_1;
  real_T cf;
  real_T sigmat;
  int32_T exitg2;
  int32_T k;
  char_T a[10];
  char_T wav[5];
  boolean_T b_b[2];
  boolean_T exitg1;
  boolean_T freqsep;
  boolean_T guard1;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  b_self = self;
  b_self->CutOff = 50.0;
  b_self->Gamma = 3.0;
  b_self->Beta = 20.0;
  for (k = 0; k < 5; k++) {
    b_self->Wavelet[k] = b_cv[k];
  }
  b_self->TimeBandwidth = rtNaN;
  b_self->SignalLength = varargin_4;
  NyquistRange_idx_1 = b_self->SignalLength / 2.0;
  NyquistRange_idx_1 = std::floor(NyquistRange_idx_1);
  b_self->SignalPad = NyquistRange_idx_1;
  b_self->VoicesPerOctave = varargin_10;
  b_self->SamplingFrequency = varargin_6;
  b_self->WaveletParameters[0] = rtNaN;
  b_self->FrequencyLimits[0] = varargin_8[0];
  b_self->WaveletParameters[1] = rtNaN;
  b_self->FrequencyLimits[1] = varargin_8[1];
  for (k = 0; k < 10; k++) {
    b_self->Boundary[k] = b[k];
  }
  NyquistRange_idx_1 = b_self->TimeBandwidth;
  guard1 = false;
  if (!std::isnan(NyquistRange_idx_1)) {
    NyquistRange_idx_1 = b_self->WaveletParameters[0];
    b_b[0] = std::isnan(NyquistRange_idx_1);
    NyquistRange_idx_1 = b_self->WaveletParameters[1];
    b_b[1] = std::isnan(NyquistRange_idx_1);
    freqsep = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k < 2)) {
      if (!b_b[k]) {
        freqsep = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    if (freqsep) {
      b_self->Beta = b_self->TimeBandwidth / b_self->Gamma;
    } else {
      guard1 = true;
    }
  } else {
    guard1 = true;
  }
  if (guard1) {
    NyquistRange_idx_1 = b_self->WaveletParameters[0];
    b_b[0] = std::isnan(NyquistRange_idx_1);
    NyquistRange_idx_1 = b_self->WaveletParameters[1];
    b_b[1] = std::isnan(NyquistRange_idx_1);
    freqsep = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k < 2)) {
      if (!b_b[k]) {
        freqsep = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    if (!freqsep) {
      NyquistRange_idx_1 = b_self->TimeBandwidth;
      if (std::isnan(NyquistRange_idx_1)) {
        b_self->Gamma = b_self->WaveletParameters[0];
        b_self->Beta = b_self->WaveletParameters[1] / b_self->Gamma;
      }
    }
  }
  for (k = 0; k < 10; k++) {
    a[k] = b_self->Boundary[k];
  }
  freqsep = false;
  k = 0;
  do {
    exitg2 = 0;
    if (k + 1 < 11) {
      if (cv[static_cast<uint8_T>(a[k]) & 127] !=
          cv[static_cast<int32_T>(cv2[k])]) {
        exitg2 = 1;
      } else {
        k++;
      }
    } else {
      freqsep = true;
      exitg2 = 1;
    }
  } while (exitg2 == 0);
  if (freqsep) {
    if (b_self->SignalLength <= 100000.0) {
      NyquistRange_idx_1 = b_self->SignalLength / 2.0;
      NyquistRange_idx_1 = std::floor(NyquistRange_idx_1);
      b_self->SignalPad = NyquistRange_idx_1;
    } else {
      NyquistRange_idx_1 = b_log2(b_self->SignalLength);
      NyquistRange_idx_1 = std::ceil(NyquistRange_idx_1);
      b_self->SignalPad = NyquistRange_idx_1;
    }
  } else {
    b_self->SignalPad = 0.0;
  }
  NyquistRange_idx_1 = b_self->FrequencyLimits[0];
  b_b[0] = std::isnan(NyquistRange_idx_1);
  NyquistRange_idx_1 = b_self->FrequencyLimits[1];
  b_b[1] = std::isnan(NyquistRange_idx_1);
  freqsep = true;
  k = 0;
  exitg1 = false;
  while ((!exitg1) && (k < 2)) {
    if (!b_b[k]) {
      freqsep = false;
      exitg1 = true;
    } else {
      k++;
    }
  }
  if (!freqsep) {
    real_T FourierFactor;
    real_T be;
    real_T cutoff;
    real_T freqrange_idx_0;
    real_T freqrange_idx_1;
    real_T fs;
    real_T ga;
    real_T omegac;
    real_T varargin_1;
    real_T varargin_3;
    char_T b_wav[5];
    freqrange_idx_0 = b_self->FrequencyLimits[0];
    freqrange_idx_1 = b_self->FrequencyLimits[1];
    NyquistRange_idx_1 = b_self->SamplingFrequency / 2.0;
    emxInit_char_T(&tmpStr, 2, &ic_emlrtRTEI, true);
    if ((freqrange_idx_1 <= 0.0) || (freqrange_idx_0 >= NyquistRange_idx_1)) {
      b_tmpStr = nullptr;
      y = nullptr;
      m = emlrtCreateCharArray(2, &iv[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &u[0]);
      emlrtAssign(&y, m);
      b_y = nullptr;
      m1 = emlrtCreateCharArray(2, &iv1[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 2, m1, &formatSpec[0]);
      emlrtAssign(&b_y, m1);
      c_y = nullptr;
      m2 = emlrtCreateDoubleScalar(NyquistRange_idx_1);
      emlrtAssign(&c_y, m2);
      emlrtAssign(&b_tmpStr, feval(y, b_y, c_y, &emlrtMCI));
      emlrt_marshallIn(length(emlrtAlias(b_tmpStr), &b_emlrtMCI),
                       "<output of length>");
      emlrt_marshallIn(emlrtAlias(b_tmpStr), "tmpStr", tmpStr);
      emlrtDestroyArray(&b_tmpStr);
    }
    emxFree_char_T(&tmpStr);
    fs = b_self->SamplingFrequency;
    ga = b_self->Gamma;
    be = b_self->Beta;
    NyquistRange_idx_1 = b_self->SignalLength;
    varargin_3 = b_self->VoicesPerOctave;
    cutoff = b_self->CutOff;
    varargin_1 = b_self->SamplingFrequency;
    for (k = 0; k < 5; k++) {
      char_T c;
      c = b_self->Wavelet[k];
      wav[k] = cv[static_cast<uint8_T>(c) & 127];
    }
    omegac = 3.1415926535897931;
    cutoff /= 100.0;
    for (int32_T d{0}; d < 5; d++) {
      b_wav[d] = wav[d];
    }
    FourierFactor =
        wavelet::internal::cwt::wavCFandSD(b_wav, ga, be, &sigmat, &cf);
    sigmat = NyquistRange_idx_1 / (sigmat * 2.0);
    k = 0;
    do {
      exitg2 = 0;
      if (k + 1 < 6) {
        if (cv4[k] != wav[k]) {
          exitg2 = 1;
        } else {
          k++;
        }
      } else {
        freqsep = true;
        exitg2 = 1;
      }
    } while (exitg2 == 0);
    if (freqsep) {
      k = 0;
    } else {
      k = -1;
    }
    if (k == 0) {
      omegac =
          wavelet::internal::cwt::getFreqFromCutoffMorse(cutoff, cf, ga, be);
    }
    NyquistRange_idx_1 = omegac / 3.1415926535897931;
    if (sigmat < NyquistRange_idx_1 * rt_powd_snf(2.0, 1.0 / varargin_3)) {
      sigmat = NyquistRange_idx_1 * rt_powd_snf(2.0, 1.0 / varargin_3);
    }
    NyquistRange_idx_1 = 1.0 / (sigmat * FourierFactor) * varargin_1;
    if (freqrange_idx_0 < NyquistRange_idx_1) {
      b_self->FrequencyLimits[0] = NyquistRange_idx_1;
      freqrange_idx_0 = b_self->FrequencyLimits[0];
    }
    if (freqrange_idx_1 > fs / 2.0) {
      b_self->FrequencyLimits[1] = fs / 2.0;
      freqrange_idx_1 = b_self->FrequencyLimits[1];
    }
    freqsep = (b_log2(freqrange_idx_1) - b_log2(freqrange_idx_0) >=
               1.0 / b_self->VoicesPerOctave);
    if (!freqsep) {
      varargin_1 = 1.0 / b_self->VoicesPerOctave;
      c_tmpStr = nullptr;
      d_y = nullptr;
      m3 = emlrtCreateCharArray(2, &iv2[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m3, &u[0]);
      emlrtAssign(&d_y, m3);
      e_y = nullptr;
      m4 = emlrtCreateCharArray(2, &iv3[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m4, &b_formatSpec[0]);
      emlrtAssign(&e_y, m4);
      f_y = nullptr;
      m5 = emlrtCreateDoubleScalar(varargin_1);
      emlrtAssign(&f_y, m5);
      emlrtAssign(&c_tmpStr, feval(d_y, e_y, f_y, &emlrtMCI));
      emlrt_marshallIn(length(emlrtAlias(c_tmpStr), &b_emlrtMCI),
                       "<output of length>");
      emxInit_char_T(&d_tmpStr, 2, &ic_emlrtRTEI, true);
      emlrt_marshallIn(emlrtAlias(c_tmpStr), "tmpStr", d_tmpStr);
      emxFree_char_T(&d_tmpStr);
      emlrtDestroyArray(&c_tmpStr);
    }
  }
  for (k = 0; k < 5; k++) {
    wav[k] = b_self->Wavelet[k];
  }
  freqsep = false;
  k = 0;
  do {
    exitg2 = 0;
    if (k + 1 < 6) {
      if (cv[static_cast<uint8_T>(wav[k]) & 127] !=
          cv[static_cast<int32_T>(cv3[k])]) {
        exitg2 = 1;
      } else {
        k++;
      }
    } else {
      freqsep = true;
      exitg2 = 1;
    }
  } while (exitg2 == 0);
  if (!freqsep) {
    b_self->CutOff = 10.0;
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  return b_self;
}

} // namespace coder
static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(tmpStr), &thisId, y);
  emlrtDestroyArray(&tmpStr);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y)
{
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location)
{
  const mxArray *pArrays[3];
  const mxArray *m;
  pArrays[0] = m1;
  pArrays[1] = m2;
  pArrays[2] = m3;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 3, &pArrays[0],
                               "feval", true, location);
}

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location)
{
  const mxArray *m;
  const mxArray *pArray;
  pArray = m1;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 1, &pArray, "length",
                               true, location);
}

//
//
namespace coder {
cwtfilterbank *cwtfilterbank_cwtfilterbank(cwtfilterbank *self,
                                           real_T varargin_4, real_T varargin_6,
                                           const real_T varargin_8[2],
                                           real_T varargin_10)
{
  static const char_T cv1[5]{'m', 'o', 'r', 's', 'e'};
  static const char_T cv2[5]{'M', 'o', 'r', 's', 'e'};
  cwtfilterbank *b_self;
  emxArray_real_T *absomega;
  emxArray_real_T *b_somega;
  emxArray_real_T *b_x;
  emxArray_real_T *b_y;
  emxArray_real_T *c_b;
  emxArray_real_T *c_y;
  emxArray_real_T *d_b;
  emxArray_real_T *d_y;
  emxArray_real_T *expnt;
  emxArray_real_T *f;
  emxArray_real_T *omega_tmp1;
  emxArray_real_T *omega_tmp2;
  emxArray_real_T *powscales;
  emxArray_real_T *psidft;
  emxArray_real_T *somega;
  emxArray_real_T *w;
  real_T N;
  real_T b;
  real_T be;
  real_T cf;
  real_T cutoff;
  real_T fo;
  real_T ga;
  int32_T b_i;
  int32_T exitg2;
  int32_T i;
  int32_T i2;
  int32_T loop_ub;
  int32_T nx;
  char_T x[5];
  boolean_T b_b[2];
  boolean_T exitg1;
  boolean_T y;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  b_self = self;
  b_self = cwtfilterbank_setProperties(b_self, varargin_4, varargin_6,
                                       varargin_8, varargin_10);
  N = b_self->SignalLength + 2.0 * b_self->SignalPad;
  b = std::trunc(N / 2.0);
  emxInit_real_T(&omega_tmp1, 2, &db_emlrtRTEI, true);
  if (std::isnan(b)) {
    i = omega_tmp1->size[0] * omega_tmp1->size[1];
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = 1;
    emxEnsureCapacity_real_T(omega_tmp1, i, &db_emlrtRTEI);
    omega_tmp1->data[0] = rtNaN;
  } else if (b < 1.0) {
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = 0;
  } else {
    i = omega_tmp1->size[0] * omega_tmp1->size[1];
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = static_cast<int32_T>(b - 1.0) + 1;
    emxEnsureCapacity_real_T(omega_tmp1, i, &db_emlrtRTEI);
    for (i = 0; i <= static_cast<int32_T>(b - 1.0); i++) {
      omega_tmp1->data[i] = static_cast<real_T>(i) + 1.0;
    }
  }
  emxInit_real_T(&omega_tmp2, 2, &eb_emlrtRTEI, true);
  i = omega_tmp2->size[0] * omega_tmp2->size[1];
  omega_tmp2->size[0] = 1;
  omega_tmp2->size[1] = omega_tmp1->size[1];
  emxEnsureCapacity_real_T(omega_tmp2, i, &eb_emlrtRTEI);
  cutoff = 6.2831853071795862 / N;
  for (i = 0; i < omega_tmp1->size[1]; i++) {
    omega_tmp2->data[i] = omega_tmp1->data[i] * cutoff;
  }
  emxFree_real_T(&omega_tmp1);
  b = std::trunc((N - 1.0) / 2.0);
  if (b < 1.0) {
    b_i = 0;
    nx = 1;
    i2 = -1;
  } else {
    b_i = static_cast<int32_T>(b) - 1;
    nx = -1;
    i2 = 0;
  }
  i = b_self->Omega->size[0] * b_self->Omega->size[1];
  b_self->Omega->size[0] = 1;
  b_self->Omega->size[1] = (omega_tmp2->size[1] + div_s32(i2 - b_i, nx)) + 2;
  emxEnsureCapacity_real_T(b_self->Omega, i, &fb_emlrtRTEI);
  b_self->Omega->data[0] = 0.0;
  loop_ub = omega_tmp2->size[1];
  for (i = 0; i < loop_ub; i++) {
    b_self->Omega->data[i + 1] = omega_tmp2->data[i];
  }
  loop_ub = div_s32(i2 - b_i, nx);
  for (i = 0; i <= loop_ub; i++) {
    b_self->Omega->data[(i + omega_tmp2->size[1]) + 1] =
        -omega_tmp2->data[b_i + nx * i];
  }
  emxFree_real_T(&omega_tmp2);
  cutoff = b_self->FrequencyLimits[0];
  b_b[0] = std::isnan(cutoff);
  cutoff = b_self->FrequencyLimits[1];
  b_b[1] = std::isnan(cutoff);
  y = true;
  i2 = 0;
  exitg1 = false;
  while ((!exitg1) && (i2 < 2)) {
    if (!b_b[i2]) {
      y = false;
      exitg1 = true;
    } else {
      i2++;
    }
  }
  if (!y) {
    real_T nv;
    real_T omegac;
    char_T c_x[5];
    cutoff = b_self->FrequencyLimits[0];
    cutoff /= b_self->SamplingFrequency;
    cutoff = cutoff * 2.0 * 3.1415926535897931;
    omegac = cutoff;
    cutoff = b_self->FrequencyLimits[1];
    cutoff /= b_self->SamplingFrequency;
    cutoff = cutoff * 2.0 * 3.1415926535897931;
    nv = b_self->VoicesPerOctave;
    N = rt_powd_snf(2.0, 1.0 / nv);
    for (int32_T d{0}; d < 5; d++) {
      c_x[d] = b_self->Wavelet[d];
    }
    wavelet::internal::cwt::wavCFandSD(c_x, b_self->Gamma, b_self->Beta, &fo,
                                       &cf);
    fo = cf / cutoff;
    emxInit_real_T(&b_y, 2, &wb_emlrtRTEI, true);
    b = nv * b_log2(cf / omegac / fo);
    if (std::isnan(b)) {
      i = b_y->size[0] * b_y->size[1];
      b_y->size[0] = 1;
      b_y->size[1] = 1;
      emxEnsureCapacity_real_T(b_y, i, &gb_emlrtRTEI);
      b_y->data[0] = rtNaN;
    } else if (b < 0.0) {
      b_y->size[0] = 1;
      b_y->size[1] = 0;
    } else {
      i = b_y->size[0] * b_y->size[1];
      b_y->size[0] = 1;
      b_y->size[1] = static_cast<int32_T>(b) + 1;
      emxEnsureCapacity_real_T(b_y, i, &gb_emlrtRTEI);
      for (i = 0; i <= static_cast<int32_T>(b); i++) {
        b_y->data[i] = i;
      }
    }
    emxInit_real_T(&c_b, 2, &xb_emlrtRTEI, true);
    i = c_b->size[0] * c_b->size[1];
    c_b->size[0] = 1;
    c_b->size[1] = b_y->size[1];
    emxEnsureCapacity_real_T(c_b, i, &o_emlrtRTEI);
    nx = b_y->size[1];
    for (i2 = 0; i2 < nx; i2++) {
      c_b->data[i2] = rt_powd_snf(N, b_y->data[i2]);
    }
    emxFree_real_T(&b_y);
    i = b_self->Scales->size[0] * b_self->Scales->size[1];
    b_self->Scales->size[0] = 1;
    b_self->Scales->size[1] = c_b->size[1];
    emxEnsureCapacity_real_T(b_self->Scales, i, &hb_emlrtRTEI);
    for (i = 0; i < c_b->size[1]; i++) {
      b_self->Scales->data[i] = fo * c_b->data[i];
    }
    emxFree_real_T(&c_b);
  } else {
    real_T nv;
    real_T omegac;
    char_T c_x[5];
    N = b_self->SignalLength;
    ga = b_self->Gamma;
    be = b_self->Beta;
    nv = b_self->VoicesPerOctave;
    cutoff = b_self->CutOff;
    for (i2 = 0; i2 < 5; i2++) {
      char_T c;
      c = b_self->Wavelet[i2];
      x[i2] = cv[static_cast<uint8_T>(c) & 127];
    }
    omegac = 3.1415926535897931;
    cutoff /= 100.0;
    for (int32_T e{0}; e < 5; e++) {
      c_x[e] = x[e];
    }
    wavelet::internal::cwt::wavCFandSD(c_x, ga, be, &fo, &cf);
    b = N / (fo * 2.0);
    y = false;
    nx = 0;
    do {
      exitg2 = 0;
      if (nx + 1 < 6) {
        if (cv1[nx] != x[nx]) {
          exitg2 = 1;
        } else {
          nx++;
        }
      } else {
        y = true;
        exitg2 = 1;
      }
    } while (exitg2 == 0);
    if (y) {
      nx = 0;
    } else {
      nx = -1;
    }
    if (nx == 0) {
      omegac =
          wavelet::internal::cwt::getFreqFromCutoffMorse(cutoff, cf, ga, be);
    }
    fo = omegac / 3.1415926535897931;
    if (b < fo * rt_powd_snf(2.0, 1.0 / nv)) {
      b = fo * rt_powd_snf(2.0, 1.0 / nv);
    }
    N = rt_powd_snf(2.0, 1.0 / nv);
    emxInit_real_T(&c_y, 2, &ac_emlrtRTEI, true);
    b = std::fmax(b_log2(b / fo), 1.0 / nv) * nv;
    if (std::isnan(b)) {
      i = c_y->size[0] * c_y->size[1];
      c_y->size[0] = 1;
      c_y->size[1] = 1;
      emxEnsureCapacity_real_T(c_y, i, &gb_emlrtRTEI);
      c_y->data[0] = rtNaN;
    } else if (b < 0.0) {
      c_y->size[0] = 1;
      c_y->size[1] = 0;
    } else {
      i = c_y->size[0] * c_y->size[1];
      c_y->size[0] = 1;
      c_y->size[1] = static_cast<int32_T>(b) + 1;
      emxEnsureCapacity_real_T(c_y, i, &gb_emlrtRTEI);
      for (i = 0; i <= static_cast<int32_T>(b); i++) {
        c_y->data[i] = i;
      }
    }
    emxInit_real_T(&d_b, 2, &dc_emlrtRTEI, true);
    i = d_b->size[0] * d_b->size[1];
    d_b->size[0] = 1;
    d_b->size[1] = c_y->size[1];
    emxEnsureCapacity_real_T(d_b, i, &o_emlrtRTEI);
    nx = c_y->size[1];
    for (i2 = 0; i2 < nx; i2++) {
      d_b->data[i2] = rt_powd_snf(N, c_y->data[i2]);
    }
    emxFree_real_T(&c_y);
    i = b_self->Scales->size[0] * b_self->Scales->size[1];
    b_self->Scales->size[0] = 1;
    b_self->Scales->size[1] = d_b->size[1];
    emxEnsureCapacity_real_T(b_self->Scales, i, &ib_emlrtRTEI);
    for (i = 0; i < d_b->size[1]; i++) {
      b_self->Scales->data[i] = fo * d_b->data[i];
    }
    emxFree_real_T(&d_b);
  }
  for (i = 0; i < 5; i++) {
    x[i] = b_self->Wavelet[i];
  }
  y = false;
  nx = 0;
  do {
    exitg2 = 0;
    if (nx + 1 < 6) {
      if (cv[static_cast<uint8_T>(x[nx]) & 127] !=
          cv[static_cast<int32_T>(cv2[nx])]) {
        exitg2 = 1;
      } else {
        nx++;
      }
    } else {
      y = true;
      exitg2 = 1;
    }
  } while (exitg2 == 0);
  emxInit_real_T(&psidft, 2, &gc_emlrtRTEI, true);
  emxInit_real_T(&f, 2, &hc_emlrtRTEI, true);
  if (y) {
    ga = b_self->Gamma;
    be = b_self->Beta;
    emxInit_real_T(&b_somega, 2, &bc_emlrtRTEI, true);
    if (b_self->Scales->size[1] == 1) {
      bsxfun(b_self->Scales, b_self->Omega, b_somega);
    } else {
      i = b_somega->size[0] * b_somega->size[1];
      b_somega->size[0] = b_self->Scales->size[1];
      b_somega->size[1] = b_self->Omega->size[1];
      emxEnsureCapacity_real_T(b_somega, i, &lb_emlrtRTEI);
      loop_ub = b_self->Omega->size[1];
      for (i = 0; i < loop_ub; i++) {
        nx = b_self->Scales->size[1];
        for (i2 = 0; i2 < nx; i2++) {
          b_somega->data[i2 + b_somega->size[0] * i] =
              b_self->Scales->data[i2] * b_self->Omega->data[i];
        }
      }
    }
    fo = std::exp(1.0 / ga * (std::log(be) - std::log(ga)));
    nx = b_somega->size[0] * b_somega->size[1];
    emxInit_real_T(&absomega, 2, &ec_emlrtRTEI, true);
    i = absomega->size[0] * absomega->size[1];
    absomega->size[0] = b_somega->size[0];
    absomega->size[1] = b_somega->size[1];
    emxEnsureCapacity_real_T(absomega, i, &n_emlrtRTEI);
    for (i2 = 0; i2 < nx; i2++) {
      absomega->data[i2] = std::abs(b_somega->data[i2]);
    }
    emxInit_real_T(&powscales, 2, &nb_emlrtRTEI, true);
    if (ga == 3.0) {
      i = powscales->size[0] * powscales->size[1];
      powscales->size[0] = absomega->size[0];
      powscales->size[1] = absomega->size[1];
      emxEnsureCapacity_real_T(powscales, i, &nb_emlrtRTEI);
      for (i = 0; i < absomega->size[0] * absomega->size[1]; i++) {
        powscales->data[i] =
            absomega->data[i] * absomega->data[i] * absomega->data[i];
      }
    } else {
      i = powscales->size[0] * powscales->size[1];
      powscales->size[0] = absomega->size[0];
      powscales->size[1] = absomega->size[1];
      emxEnsureCapacity_real_T(powscales, i, &o_emlrtRTEI);
      nx = absomega->size[0] * absomega->size[1];
      for (i2 = 0; i2 < nx; i2++) {
        powscales->data[i2] = rt_powd_snf(absomega->data[i2], ga);
      }
    }
    nx = absomega->size[0] * absomega->size[1];
    for (i2 = 0; i2 < nx; i2++) {
      absomega->data[i2] = std::log(absomega->data[i2]);
    }
    b = 2.0 * std::exp(-be * std::log(fo) + rt_powd_snf(fo, ga));
    emxInit_real_T(&b_x, 2, &ob_emlrtRTEI, true);
    if ((absomega->size[0] == powscales->size[0]) &&
        (absomega->size[1] == powscales->size[1])) {
      i = b_x->size[0] * b_x->size[1];
      b_x->size[0] = absomega->size[0];
      b_x->size[1] = absomega->size[1];
      emxEnsureCapacity_real_T(b_x, i, &ob_emlrtRTEI);
      for (i = 0; i < absomega->size[0] * absomega->size[1]; i++) {
        b_x->data[i] = be * absomega->data[i] - powscales->data[i];
      }
    } else {
      binary_expand_op_2(b_x, be, absomega, powscales);
    }
    emxFree_real_T(&powscales);
    emxFree_real_T(&absomega);
    nx = b_x->size[0] * b_x->size[1];
    for (i2 = 0; i2 < nx; i2++) {
      b_x->data[i2] = std::exp(b_x->data[i2]);
    }
    if ((b_x->size[0] == b_somega->size[0]) &&
        (b_x->size[1] == b_somega->size[1])) {
      i = psidft->size[0] * psidft->size[1];
      psidft->size[0] = b_x->size[0];
      psidft->size[1] = b_x->size[1];
      emxEnsureCapacity_real_T(psidft, i, &rb_emlrtRTEI);
      for (i = 0; i < b_x->size[0] * b_x->size[1]; i++) {
        psidft->data[i] =
            b * b_x->data[i] * static_cast<real_T>(b_somega->data[i] > 0.0);
      }
    } else {
      binary_expand_op_1(psidft, b, b_x, b_somega);
    }
    emxFree_real_T(&b_x);
    emxFree_real_T(&b_somega);
    i = f->size[0] * f->size[1];
    f->size[0] = 1;
    f->size[1] = b_self->Scales->size[1];
    emxEnsureCapacity_real_T(f, i, &sb_emlrtRTEI);
    for (i = 0; i < b_self->Scales->size[1]; i++) {
      f->data[i] = fo / b_self->Scales->data[i] / 6.2831853071795862;
    }
  } else {
    emxInit_real_T(&somega, 2, &yb_emlrtRTEI, true);
    if (b_self->Scales->size[1] == 1) {
      bsxfun(b_self->Scales, b_self->Omega, somega);
    } else {
      i = somega->size[0] * somega->size[1];
      somega->size[0] = b_self->Scales->size[1];
      somega->size[1] = b_self->Omega->size[1];
      emxEnsureCapacity_real_T(somega, i, &jb_emlrtRTEI);
      loop_ub = b_self->Omega->size[1];
      for (i = 0; i < loop_ub; i++) {
        nx = b_self->Scales->size[1];
        for (i2 = 0; i2 < nx; i2++) {
          somega->data[i2 + somega->size[0] * i] =
              b_self->Scales->data[i2] * b_self->Omega->data[i];
        }
      }
    }
    emxInit_real_T(&w, 2, &kb_emlrtRTEI, true);
    i = w->size[0] * w->size[1];
    w->size[0] = somega->size[0];
    w->size[1] = somega->size[1];
    emxEnsureCapacity_real_T(w, i, &kb_emlrtRTEI);
    for (i = 0; i < somega->size[0] * somega->size[1]; i++) {
      w->data[i] = (somega->data[i] - 5.0) / 0.6;
    }
    emxFree_real_T(&somega);
    emxInit_real_T(&expnt, 2, &cc_emlrtRTEI, true);
    i = expnt->size[0] * expnt->size[1];
    expnt->size[0] = w->size[0];
    expnt->size[1] = w->size[1];
    emxEnsureCapacity_real_T(expnt, i, &mb_emlrtRTEI);
    for (i = 0; i < w->size[0] * w->size[1]; i++) {
      expnt->data[i] = -1.0 / (1.0 - w->data[i] * w->data[i]);
    }
    nx = expnt->size[0] * expnt->size[1];
    for (i2 = 0; i2 < nx; i2++) {
      expnt->data[i2] = std::exp(expnt->data[i2]);
    }
    nx = w->size[0] * w->size[1];
    emxInit_real_T(&d_y, 2, &fc_emlrtRTEI, true);
    i = d_y->size[0] * d_y->size[1];
    d_y->size[0] = w->size[0];
    d_y->size[1] = w->size[1];
    emxEnsureCapacity_real_T(d_y, i, &n_emlrtRTEI);
    for (i2 = 0; i2 < nx; i2++) {
      d_y->data[i2] = std::abs(w->data[i2]);
    }
    emxFree_real_T(&w);
    if ((expnt->size[0] == d_y->size[0]) && (expnt->size[1] == d_y->size[1])) {
      i = psidft->size[0] * psidft->size[1];
      psidft->size[0] = expnt->size[0];
      psidft->size[1] = expnt->size[1];
      emxEnsureCapacity_real_T(psidft, i, &pb_emlrtRTEI);
      for (i = 0; i < expnt->size[0] * expnt->size[1]; i++) {
        psidft->data[i] =
            5.43656365691809 * expnt->data[i] *
            static_cast<real_T>(d_y->data[i] < 0.99999999999999978);
      }
    } else {
      binary_expand_op_3(psidft, expnt, d_y);
    }
    emxFree_real_T(&d_y);
    emxFree_real_T(&expnt);
    nx = psidft->size[0] * psidft->size[1] - 1;
    for (b_i = 0; b_i <= nx; b_i++) {
      if (std::isnan(psidft->data[b_i])) {
        psidft->data[b_i] = 0.0;
      }
    }
    i = f->size[0] * f->size[1];
    f->size[0] = 1;
    f->size[1] = b_self->Scales->size[1];
    emxEnsureCapacity_real_T(f, i, &qb_emlrtRTEI);
    for (i = 0; i < b_self->Scales->size[1]; i++) {
      f->data[i] = 0.79577471545947676 / b_self->Scales->data[i];
    }
  }
  i = f->size[0] * f->size[1];
  f->size[0] = 1;
  emxEnsureCapacity_real_T(f, i, &tb_emlrtRTEI);
  i = f->size[1] - 1;
  for (i2 = 0; i2 <= i; i2++) {
    f->data[i2] *= b_self->SamplingFrequency;
  }
  i = b_self->PsiDFT->size[0] * b_self->PsiDFT->size[1];
  b_self->PsiDFT->size[0] = psidft->size[0];
  b_self->PsiDFT->size[1] = psidft->size[1];
  emxEnsureCapacity_real_T(b_self->PsiDFT, i, &ub_emlrtRTEI);
  for (i = 0; i < psidft->size[0] * psidft->size[1]; i++) {
    b_self->PsiDFT->data[i] = psidft->data[i];
  }
  emxFree_real_T(&psidft);
  i = b_self->WaveletCenterFrequencies->size[0];
  b_self->WaveletCenterFrequencies->size[0] = f->size[1];
  emxEnsureCapacity_real_T(b_self->WaveletCenterFrequencies, i, &vb_emlrtRTEI);
  for (i = 0; i < f->size[1]; i++) {
    b_self->WaveletCenterFrequencies->data[i] = f->data[i];
  }
  emxFree_real_T(&f);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  return b_self;
}

} // namespace coder

// End of code generation (cwtfilterbank.cu)

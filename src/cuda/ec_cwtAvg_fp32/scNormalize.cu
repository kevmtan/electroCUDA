#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// scNormalize.cu
//
// Code generation for function 'scNormalize'
//

// Include files
#include "scNormalize.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo qf_emlrtRTEI{
    12,                                                               // lineNo
    5,                                                                // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo rf_emlrtRTEI{
    12,            // lineNo
    12,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo sf_emlrtRTEI{
    22,            // lineNo
    32,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo tf_emlrtRTEI{
    12,            // lineNo
    1,             // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo uf_emlrtRTEI{
    26,            // lineNo
    43,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo vf_emlrtRTEI{
    26,            // lineNo
    37,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

// Function Declarations
static real32_T rt_hypotf_snf(real32_T u0, real32_T u1);

static __global__ void scNormalize_kernel46(const real32_T c1,
                                            emxArray_real32_T c);

static __global__ void
scNormalize_kernel47(const real32_T c1, emxArray_real32_T c, int32_T z_dim0);

// Function Definitions
static real32_T rt_hypotf_snf(real32_T u0, real32_T u1)
{
  real32_T a;
  real32_T b;
  real32_T y;
  a = std::abs(u0);
  b = std::abs(u1);
  if (a < b) {
    a /= b;
    y = b * std::sqrt(a * a + 1.0F);
  } else if (a > b) {
    b /= a;
    y = a * std::sqrt(b * b + 1.0F);
  } else if (std::isnan(b)) {
    y = rtNaNF;
  } else {
    y = a * 1.41421354F;
  }
  return y;
}

static __global__
    __launch_bounds__(32, 1) void scNormalize_kernel46(const real32_T c1,
                                                       emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = c1;
  }
}

static __global__ __launch_bounds__(32, 1) void scNormalize_kernel47(
    const real32_T c1, emxArray_real32_T c, int32_T z_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[z_dim0 - 1] = c1;
  }
}

//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
void scNormalize(emxArray_creal32_T *cfs, real_T cpsi,
                 const emxArray_real32_T *scales, real32_T normfac)
{
  emxArray_real32_T gpu_c;
  emxArray_real32_T gpu_z;
  emxArray_real32_T *a;
  emxArray_real32_T *abscfssq;
  emxArray_real32_T *cpu_c;
  emxArray_real32_T *cpu_z;
  emxArray_real32_T *y;
  emxArray_real_T *c;
  emxArray_uint32_T *b_y;
  int32_T nx;
  int32_T u1;
  int32_T vstride;
  real32_T b_c1;
  real32_T cpu_b;
  real32_T *gpu_b;
  boolean_T b_outdatedOnCpu;
  boolean_T z_needsGpuEnsureCapacity;
  boolean_T z_outdatedOnGpu;
  gpuEmxReset_real32_T(&gpu_c);
  checkCudaError(mwCudaMalloc(&gpu_b, 4UL), __FILE__, __LINE__);
  gpuEmxReset_real32_T(&gpu_z);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nx = cfs->size[0] * cfs->size[1];
  emxInit_real32_T(&y, 2, &rf_emlrtRTEI, true);
  vstride = y->size[0] * y->size[1];
  y->size[0] = cfs->size[0];
  y->size[1] = cfs->size[1];
  emxEnsureCapacity_real32_T(y, vstride, &n_emlrtRTEI);
  for (int32_T k{0}; k < nx; k++) {
    y->data[k] = rt_hypotf_snf(cfs->data[k].re, cfs->data[k].im);
  }
  emxInit_real32_T(&a, 2, &sf_emlrtRTEI, true);
  vstride = a->size[0] * a->size[1];
  a->size[0] = y->size[0];
  a->size[1] = y->size[1];
  emxEnsureCapacity_real32_T(a, vstride, &o_emlrtRTEI);
  nx = y->size[0] * y->size[1];
  for (int32_T k{0}; k < nx; k++) {
    a->data[k] = y->data[k] * y->data[k];
  }
  emxFree_real32_T(&y);
  emxInit_real32_T(&abscfssq, 2, &tf_emlrtRTEI, true);
  vstride = abscfssq->size[0] * abscfssq->size[1];
  nx = scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (scales->size[1] == 1) {
    abscfssq->size[0] = a->size[0];
  } else if (a->size[0] == 1) {
    abscfssq->size[0] = scales->size[1];
  } else if (a->size[0] == scales->size[1]) {
    abscfssq->size[0] = a->size[0];
  } else {
    abscfssq->size[0] = u1;
  }
  abscfssq->size[1] = a->size[1];
  emxEnsureCapacity_real32_T(abscfssq, vstride, &i_emlrtRTEI);
  nx = scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (scales->size[1] == 1) {
    u1 = a->size[0];
  } else if (a->size[0] == 1) {
    u1 = scales->size[1];
  } else if (a->size[0] == scales->size[1]) {
    u1 = a->size[0];
  }
  if ((u1 != 0) && (a->size[1] != 0)) {
    int32_T acoef;
    int32_T bcoef;
    nx = (a->size[1] != 1);
    u1 = a->size[1] - 1;
    acoef = (a->size[0] != 1);
    bcoef = (scales->size[1] != 1);
    for (int32_T k{0}; k <= u1; k++) {
      int32_T i1;
      vstride = nx * k;
      i1 = abscfssq->size[0] - 1;
      for (int32_T b_k{0}; b_k <= i1; b_k++) {
        abscfssq->data[b_k + abscfssq->size[0] * k] =
            a->data[acoef * b_k + a->size[0] * vstride] /
            scales->data[bcoef * b_k];
      }
    }
  }
  emxFree_real32_T(&a);
  emxInit_uint32_T(&b_y, 2, &uf_emlrtRTEI, true);
  if (abscfssq->size[1] < 1) {
    b_y->size[0] = 1;
    b_y->size[1] = 0;
  } else {
    uint32_T u;
    u = static_cast<uint32_T>(abscfssq->size[1]);
    vstride = b_y->size[0] * b_y->size[1];
    b_y->size[0] = 1;
    b_y->size[1] = abscfssq->size[1];
    emxEnsureCapacity_uint32_T(b_y, vstride, &gb_emlrtRTEI);
    for (vstride = 0; vstride < static_cast<int32_T>(u); vstride++) {
      b_y->data[vstride] = static_cast<uint32_T>(vstride) + 1U;
    }
  }
  emxInit_real32_T(&cpu_z, 1, &vf_emlrtRTEI, true);
  z_outdatedOnGpu = false;
  vstride = cpu_z->size[0];
  cpu_z->size[0] = abscfssq->size[0];
  emxEnsureCapacity_real32_T(cpu_z, vstride, &qf_emlrtRTEI);
  z_needsGpuEnsureCapacity = true;
  for (vstride = 0; vstride < abscfssq->size[0]; vstride++) {
    cpu_z->data[vstride] = 0.0F;
    z_outdatedOnGpu = true;
  }
  if (abscfssq->size[1] > 1) {
    emxInit_real_T(&c, 1, &t_emlrtRTEI, true);
    if (b_y->size[1] == 1) {
      nx = abscfssq->size[1];
      vstride = c->size[0];
      c->size[0] = abscfssq->size[1];
      emxEnsureCapacity_real_T(c, vstride, &t_emlrtRTEI);
      for (vstride = 0; vstride < nx; vstride++) {
        c->data[vstride] = b_y->data[0];
      }
      real_T c1;
      c1 = 0.5 * static_cast<real_T>(b_y->data[0]);
      c->data[0] = c1;
      c->data[abscfssq->size[1] - 1] = c1;
    } else {
      vstride = c->size[0];
      c->size[0] = abscfssq->size[1];
      emxEnsureCapacity_real_T(c, vstride, &s_emlrtRTEI);
      c->data[0] =
          0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[1]) -
                                    static_cast<int32_T>(b_y->data[0]));
      nx = abscfssq->size[1];
      for (int32_T k{0}; k <= nx - 3; k++) {
        c->data[k + 1] =
            0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[k + 2]) -
                                      static_cast<int32_T>(b_y->data[k]));
      }
      c->data[abscfssq->size[1] - 1] =
          0.5 * static_cast<real_T>(
                    static_cast<int32_T>(b_y->data[abscfssq->size[1] - 1]) -
                    static_cast<int32_T>(b_y->data[abscfssq->size[1] - 2]));
    }
    vstride = abscfssq->size[0];
    if (abscfssq->size[0] != 0) {
      int64_T b;
      nx = abscfssq->size[0];
      std::memset(&cpu_z->data[0], 0,
                  static_cast<uint32_T>(nx) * sizeof(real32_T));
      z_outdatedOnGpu = true;
      nx = 0;
      u1 = abscfssq->size[0] * (abscfssq->size[1] - 1) + 1;
      b = computeEndIdx(1L, static_cast<int64_T>(u1),
                        static_cast<int64_T>(vstride));
      for (int64_T iac{0L}; iac <= b; iac++) {
        int64_T d;
        d = computeEndIdx(
            static_cast<int64_T>(static_cast<int32_T>(iac * vstride + 1L)),
            static_cast<int64_T>(
                (static_cast<int32_T>(iac * vstride + 1L) + vstride) - 1),
            1L);
        for (int64_T ia{0L}; ia <= d; ia++) {
          cpu_z->data[static_cast<int32_T>(ia)] +=
              abscfssq
                  ->data[static_cast<int32_T>(
                             static_cast<int32_T>(iac * vstride + 1L) + ia) -
                         1] *
              static_cast<real32_T>(c->data[nx]);
        }
        nx++;
      }
    }
    emxFree_real_T(&c);
  }
  emxFree_uint32_T(&b_y);
  cpu_b = 0.0F;
  b_outdatedOnCpu = false;
  if (cpu_z->size[0] <= 1) {
    if ((scales->size[1] == 1) &&
        (std::isinf(scales->data[0]) || std::isnan(scales->data[0]))) {
      cpu_b = rtNaNF;
    }
  } else {
    real32_T alpha1;
    boolean_T c_needsGpuEnsureCapacity;
    emxInit_real32_T(&cpu_c, 1, &t_emlrtRTEI, true);
    if (scales->size[1] == 1) {
      nx = cpu_z->size[0];
      b_outdatedOnCpu = false;
      vstride = cpu_c->size[0];
      cpu_c->size[0] = cpu_z->size[0];
      emxEnsureCapacity_real32_T(cpu_c, vstride, &t_emlrtRTEI);
      for (vstride = 0; vstride < nx; vstride++) {
        cpu_c->data[vstride] = scales->data[0];
        b_outdatedOnCpu = true;
      }
      b_c1 = 0.5F * scales->data[0];
      gpuEmxEnsureCapacity_real32_T(cpu_c, &gpu_c, !b_outdatedOnCpu);
      if (b_outdatedOnCpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_c, cpu_c);
      }
      scNormalize_kernel46<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b_c1,
                                                                    gpu_c);
      gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
      z_needsGpuEnsureCapacity = false;
      c_needsGpuEnsureCapacity = false;
      if (z_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_z, cpu_z);
      }
      z_outdatedOnGpu = false;
      scNormalize_kernel47<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          b_c1, gpu_c, cpu_z->size[0U]);
      b_outdatedOnCpu = false;
    } else {
      vstride = cpu_c->size[0];
      cpu_c->size[0] = cpu_z->size[0];
      emxEnsureCapacity_real32_T(cpu_c, vstride, &s_emlrtRTEI);
      cpu_c->data[0] = 0.5F * (scales->data[1] - scales->data[0]);
      nx = cpu_z->size[0];
      for (int32_T k{0}; k <= nx - 3; k++) {
        cpu_c->data[k + 1] = 0.5F * (scales->data[k + 2] - scales->data[k]);
      }
      cpu_c->data[cpu_z->size[0] - 1] =
          0.5F *
          (scales->data[cpu_z->size[0] - 1] - scales->data[cpu_z->size[0] - 2]);
      b_outdatedOnCpu = true;
      c_needsGpuEnsureCapacity = true;
    }
    b_c1 = 0.0F;
    alpha1 = 1.0F;
    if (z_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
    }
    if (c_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_c, &gpu_c, !b_outdatedOnCpu);
    }
    if (z_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_z, cpu_z);
    }
    if (b_outdatedOnCpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_c, cpu_c);
    }
    checkCudaError(hipMemcpy(gpu_b, &cpu_b, 4UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
    cublasCheck(hipblasSgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                            cpu_z->size[0], 1, (float *)&alpha1,
                            (float *)&gpu_z.data[0], cpu_z->size[0],
                            (float *)&gpu_c.data[0], 1, (float *)&b_c1,
                            (float *)gpu_b, 1),
                __FILE__, __LINE__);
    b_outdatedOnCpu = true;
    emxFree_real32_T(&cpu_c);
  }
  emxFree_real32_T(&cpu_z);
  if (!(normfac == 0.0F)) {
    if (b_outdatedOnCpu) {
      checkCudaError(hipMemcpy(&cpu_b, gpu_b, 4UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    b_c1 = std::sqrt(normfac /
                     (static_cast<real32_T>(
                          2.0 / cpsi / static_cast<real_T>(abscfssq->size[1])) *
                      cpu_b));
    for (vstride = 0; vstride < cfs->size[1]; vstride++) {
      for (nx = 0; nx < cfs->size[0]; nx++) {
        cfs->data[nx + cfs->size[0] * vstride].re *= b_c1;
        cfs->data[nx + cfs->size[0] * vstride].im *= b_c1;
      }
    }
  }
  emxFree_real32_T(&abscfssq);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_z);
  checkCudaError(mwCudaFree(gpu_b), __FILE__, __LINE__);
  gpuEmxFree_real32_T(&gpu_c);
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (scNormalize.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_cwtAvg.cu
//
// Code generation for function 'ec_cwtAvg'
//

// Include files
#include "ec_cwtAvg.h"
#include "cwtfilterbank.h"
#include "div.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "numCpsi.h"
#include "resample.h"
#include "rt_nonfinite.h"
#include "scNormalize.h"
#include "vvarstd.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hipfft/hipfft.h"
#include "hip/hip_math_constants.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    26,                                                    // lineNo
    1,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    1,                                                     // lineNo
    22,                                                    // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    230,             // lineNo
    1,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    102,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo
    e_emlrtRTEI{
        105,      // lineNo
        20,       // colNo
        "varstd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
        "varstd.m" // pName
    };

static emlrtRTEInfo f_emlrtRTEI{
    135,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    143,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "cuFFTNDCallback", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "cuFFTNDCallback.p" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    138,  // lineNo
    19,   // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    138,  // lineNo
    6,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    236,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    35,                                                    // lineNo
    28,                                                    // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    298,             // lineNo
    18,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    297,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    35,                                                    // lineNo
    5,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    41,                                                    // lineNo
    9,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    39,                                                    // lineNo
    9,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    24,                                                    // lineNo
    1,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    287,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    151,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo bb_emlrtRTEI{
    293,             // lineNo
    14,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo cb_emlrtRTEI{
    39,                                                    // lineNo
    17,                                                    // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

// Function Declarations
static __global__ void ec_cwtAvg_kernel1(const int32_T ch,
                                         const emxArray_real32_T x,
                                         const int32_T b, emxArray_real32_T xv,
                                         emxArray_real32_T b_x, int32_T x_dim0);

static __global__ void ec_cwtAvg_kernel10(const emxArray_real32_T Scales,
                                          const emxArray_real32_T y,
                                          const int32_T b_y, const int32_T c_y,
                                          emxArray_real32_T abswt2S,
                                          int32_T abswt2S_dim0, int32_T y_dim0);

static __global__ void ec_cwtAvg_kernel11(const emxArray_real32_T abswt2S,
                                          const int32_T b,
                                          const int32_T b_abswt2S,
                                          emxArray_real32_T y, int32_T y_dim0,
                                          int32_T abswt2S_dim0);

static __global__ void ec_cwtAvg_kernel12(const real32_T c1,
                                          emxArray_real32_T savgpTMP);

static __global__ void ec_cwtAvg_kernel13(const emxArray_real32_T savgpTMP,
                                          const int32_T b_savgpTMP,
                                          emxArray_real32_T xCh);

static __global__ void ec_cwtAvg_kernel14(const int32_T abswt2S,
                                          emxArray_real32_T savgpTMP);

static __global__ void ec_cwtAvg_kernel15(const int32_T abswt2S,
                                          emxArray_real32_T savgpTMP);

static __global__ void ec_cwtAvg_kernel16(const emxArray_real32_T Scales,
                                          const int32_T b, emxArray_real32_T c);

static __global__ void
ec_cwtAvg_kernel17(const int32_T acoef, const real32_T c1, emxArray_real32_T c);

static __global__ void ec_cwtAvg_kernel18(const emxArray_real32_T Scales,
                                          emxArray_real32_T c);

static __global__ void ec_cwtAvg_kernel19(const emxArray_real32_T Scales,
                                          const int32_T b, emxArray_real32_T c);

static __global__ void ec_cwtAvg_kernel2(const int32_T ch,
                                         const emxArray_real32_T x,
                                         const int32_T b, emxArray_real32_T b_x,
                                         int32_T x_dim0);

static __global__ void ec_cwtAvg_kernel20(const emxArray_real32_T Scales,
                                          const int32_T acoef,
                                          emxArray_real32_T c);

static __global__ void ec_cwtAvg_kernel21(const real32_T cfsposdft_re,
                                          const emxArray_creal32_T cfsposdft,
                                          const int32_T b_cfsposdft,
                                          emxArray_creal32_T cfs);

static __global__ void ec_cwtAvg_kernel22(const emxArray_real32_T xv,
                                          const int32_T b_xv,
                                          emxArray_creal32_T xposdft);

static __global__ void ec_cwtAvg_kernel23(const int32_T b,
                                          emxArray_creal32_T xposdft,
                                          uint32_T xSize_dim1);

static __global__ void ec_cwtAvg_kernel3(const int32_T acoef, const int32_T b,
                                         emxArray_real32_T x);

static __global__ void ec_cwtAvg_kernel4(const emxArray_real32_T x,
                                         const int32_T b, emxArray_real32_T xv);

static __global__ void
ec_cwtAvg_kernel5(const emxArray_creal32_T xposdft,
                  const emxArray_real32_T psihat, const int32_T nd2,
                  const int32_T acoef, const int32_T cfsposdft,
                  const int32_T i3, emxArray_creal32_T b_cfsposdft,
                  int32_T psihat_dim0, int32_T cfsposdft_dim0);

static __global__ void ec_cwtAvg_kernel6(const int32_T xSize,
                                         emxArray_creal32_T cfs);

static __global__ void ec_cwtAvg_kernel7(const int32_T ch,
                                         const emxArray_real32_T x,
                                         const int32_T b_x,
                                         emxArray_real32_T c_x, int32_T x_dim0);

static __global__ void ec_cwtAvg_kernel8(const emxArray_creal32_T cfs,
                                         const int32_T b, emxArray_real32_T y);

static __global__ void ec_cwtAvg_kernel9(const emxArray_real32_T y,
                                         const int32_T b,
                                         emxArray_real32_T b_y);

static void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu,
                                           boolean_T needsCopy);

static void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu);

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu);

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu);

static void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel1(
    const int32_T ch, const emxArray_real32_T x, const int32_T b,
    emxArray_real32_T xv, emxArray_real32_T b_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    real32_T xtmp;
    j2 = static_cast<int32_T>(idx);
    xtmp = x.data[j2 + x_dim0 * ch];
    b_x.data[j2] = xtmp;
    xv.data[j2] = xtmp;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel10(
    const emxArray_real32_T Scales, const emxArray_real32_T y,
    const int32_T b_y, const int32_T c_y, emxArray_real32_T abswt2S,
    int32_T abswt2S_dim0, int32_T y_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_y) + 1UL) * (static_cast<uint64_T>(b_y) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T j2;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_y) + 1UL));
    j2 = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                              (static_cast<uint64_T>(b_y) + 1UL));
    abswt2S.data[bcoef + abswt2S_dim0 * j2] =
        y.data[bcoef + y_dim0 * j2] / Scales.data[bcoef];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel11(
    const emxArray_real32_T abswt2S, const int32_T b, const int32_T b_abswt2S,
    emxArray_real32_T y, int32_T y_dim0, int32_T abswt2S_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_abswt2S) + 1UL) *
                (static_cast<uint64_T>(b) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T j2;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    j2 = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                              (static_cast<uint64_T>(b) + 1UL));
    y.data[bcoef + y_dim0 * j2] = abswt2S.data[bcoef + abswt2S_dim0 * j2];
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_cwtAvg_kernel12(const real32_T c1,
                                                     emxArray_real32_T savgpTMP)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    savgpTMP.data[0] = c1;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel13(
    const emxArray_real32_T savgpTMP, const int32_T b_savgpTMP,
    emxArray_real32_T xCh)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_savgpTMP);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xCh.data[j2] = savgpTMP.data[j2];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel14(
    const int32_T abswt2S, emxArray_real32_T savgpTMP)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(abswt2S);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    savgpTMP.data[j2] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel15(
    const int32_T abswt2S, emxArray_real32_T savgpTMP)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(abswt2S);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    savgpTMP.data[j2] = HIP_NAN_F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel16(
    const emxArray_real32_T Scales, const int32_T b, emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    c.data[j2] = Scales.data[0];
  }
}

static __global__ __launch_bounds__(32, 1) void ec_cwtAvg_kernel17(
    const int32_T acoef, const real32_T c1, emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = c1;
    c.data[acoef - 1] = c1;
  }
}

static __global__ __launch_bounds__(32, 1) void ec_cwtAvg_kernel18(
    const emxArray_real32_T Scales, emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = 0.5F * (Scales.data[1] - Scales.data[0]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel19(
    const emxArray_real32_T Scales, const int32_T b, emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c.data[k + 1] = 0.5F * (Scales.data[k + 2] - Scales.data[k]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel2(
    const int32_T ch, const emxArray_real32_T x, const int32_T b,
    emxArray_real32_T b_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    b_x.data[j2] = x.data[j2 + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(32, 1) void ec_cwtAvg_kernel20(
    const emxArray_real32_T Scales, const int32_T acoef, emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[acoef - 1] =
        0.5F * (Scales.data[acoef - 1] - Scales.data[acoef - 2]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel21(
    const real32_T cfsposdft_re, const emxArray_creal32_T cfsposdft,
    const int32_T b_cfsposdft, emxArray_creal32_T cfs)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_cfsposdft);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    real32_T ai;
    real32_T xtmp;
    j2 = static_cast<int32_T>(idx);
    xtmp = cfsposdft.data[j2].re;
    ai = cfsposdft.data[j2].im;
    if (ai == 0.0F) {
      cfs.data[j2].re = xtmp / cfsposdft_re;
      cfs.data[j2].im = 0.0F;
    } else if (xtmp == 0.0F) {
      cfs.data[j2].re = 0.0F;
      cfs.data[j2].im = ai / cfsposdft_re;
    } else {
      cfs.data[j2].re = xtmp / cfsposdft_re;
      cfs.data[j2].im = ai / cfsposdft_re;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel22(
    const emxArray_real32_T xv, const int32_T b_xv, emxArray_creal32_T xposdft)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_xv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xposdft.data[j2].re = xv.data[j2];
    xposdft.data[j2].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel23(
    const int32_T b, emxArray_creal32_T xposdft, uint32_T xSize_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    bcoef = static_cast<int32_T>(idx);
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - bcoef) - 1].re =
        xposdft.data[bcoef + 1].re;
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - bcoef) - 1].im =
        -xposdft.data[bcoef + 1].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel3(
    const int32_T acoef, const int32_T b, emxArray_real32_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T j2;
    real32_T xtmp;
    bcoef = static_cast<int32_T>(idx);
    j2 = acoef - bcoef;
    xtmp = x.data[bcoef];
    x.data[bcoef] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel4(
    const emxArray_real32_T x, const int32_T b, emxArray_real32_T xv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xv.data[j2] = x.data[j2];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel5(
    const emxArray_creal32_T xposdft, const emxArray_real32_T psihat,
    const int32_T nd2, const int32_T acoef, const int32_T cfsposdft,
    const int32_T i3, emxArray_creal32_T b_cfsposdft, int32_T psihat_dim0,
    int32_T cfsposdft_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(i3) + 1UL) *
                (static_cast<uint64_T>(cfsposdft) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    int32_T bcoef;
    int32_T j2;
    int32_T k;
    int32_T varargin_2;
    b_k = static_cast<int32_T>(idx % (static_cast<uint64_T>(cfsposdft) + 1UL));
    k = static_cast<int32_T>((idx - static_cast<uint64_T>(b_k)) /
                             (static_cast<uint64_T>(cfsposdft) + 1UL));
    varargin_2 = acoef * k + 1;
    j2 = nd2 * k + 1;
    bcoef = static_cast<int32_T>(psihat_dim0 != 1);
    b_cfsposdft.data[b_k + cfsposdft_dim0 * k].re =
        psihat.data[bcoef * b_k + psihat_dim0 * (j2 - 1)] *
        xposdft.data[varargin_2 - 1].re;
    b_cfsposdft.data[b_k + cfsposdft_dim0 * k].im =
        psihat.data[bcoef * b_k + psihat_dim0 * (j2 - 1)] *
        xposdft.data[varargin_2 - 1].im;
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_cwtAvg_kernel6(const int32_T xSize,
                                                      emxArray_creal32_T cfs)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(xSize);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    cfs.data[j2].re = 0.0F;
    cfs.data[j2].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel7(
    const int32_T ch, const emxArray_real32_T x, const int32_T b_x,
    emxArray_real32_T c_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    c_x.data[j2] = x.data[j2 + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel8(
    const emxArray_creal32_T cfs, const int32_T b, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypotf(cfs.data[k].re, cfs.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel9(
    const emxArray_real32_T y, const int32_T b, emxArray_real32_T b_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_y.data[k] = y.data[k] * y.data[k];
  }
}

static void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu,
                                           boolean_T needsCopy)
{
  creal32_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(creal32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(creal32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
}

static void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_creal32_T));
}

//
// function [y,freqs] = ec_cwtAvg(x,fs,fLims,fVoices,ds)
void ec_cwtAvg(const emxArray_real32_T *cpu_x, real_T fs, const real_T fLims[2],
               real_T fVoices, const real_T ds[2], emxArray_cell_wrap_0 *y,
               emxArray_real_T *freqs)
{
  coder::cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal32_T gpu_cfs;
  emxArray_creal32_T gpu_cfsposdft;
  emxArray_creal32_T gpu_xposdft;
  emxArray_creal32_T *cpu_cfs;
  emxArray_creal32_T *cpu_cfsposdft;
  emxArray_creal32_T *cpu_xposdft;
  emxArray_real32_T b_gpu_x;
  emxArray_real32_T b_gpu_y;
  emxArray_real32_T c_gpu_x;
  emxArray_real32_T c_gpu_y;
  emxArray_real32_T d_gpu_x;
  emxArray_real32_T gpu_Scales;
  emxArray_real32_T gpu_abswt2S;
  emxArray_real32_T gpu_c;
  emxArray_real32_T gpu_psihat;
  emxArray_real32_T gpu_r;
  emxArray_real32_T gpu_savgpTMP;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_xCh;
  emxArray_real32_T gpu_xv;
  emxArray_real32_T gpu_y;
  emxArray_real32_T *b_cpu_x;
  emxArray_real32_T *b_cpu_y;
  emxArray_real32_T *c_cpu_x;
  emxArray_real32_T *c_cpu_y;
  emxArray_real32_T *cpu_Scales;
  emxArray_real32_T *cpu_abswt2S;
  emxArray_real32_T *cpu_c;
  emxArray_real32_T *cpu_psihat;
  emxArray_real32_T *cpu_r;
  emxArray_real32_T *cpu_savgpTMP;
  emxArray_real32_T *cpu_xCh;
  emxArray_real32_T *cpu_xv;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *d_cpu_x;
  int32_T i;
  int32_T i1;
  boolean_T abswt2S_outdatedOnCpu;
  boolean_T doDownsample;
  boolean_T r_outdatedOnCpu;
  boolean_T r_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T xposdft_needsGpuEnsureCapacity;
  boolean_T xv_outdatedOnCpu;
  gpuEmxReset_real32_T(&d_gpu_x);
  gpuEmxReset_real32_T(&b_gpu_x);
  gpuEmxReset_real32_T(&gpu_r);
  gpuEmxReset_real32_T(&gpu_xCh);
  gpuEmxReset_real32_T(&gpu_c);
  gpuEmxReset_real32_T(&c_gpu_y);
  gpuEmxReset_real32_T(&gpu_abswt2S);
  gpuEmxReset_real32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&gpu_y);
  gpuEmxReset_creal32_T(&gpu_cfs);
  gpuEmxReset_real32_T(&c_gpu_x);
  gpuEmxReset_creal32_T(&gpu_cfsposdft);
  gpuEmxReset_creal32_T(&gpu_xposdft);
  gpuEmxReset_real32_T(&gpu_xv);
  gpuEmxReset_real32_T(&gpu_psihat);
  gpuEmxReset_real32_T(&gpu_Scales);
  gpuEmxReset_real32_T(&gpu_savgpTMP);
  gpuEmxReset_real32_T(&gpu_x);
  abswt2S_outdatedOnCpu = false;
  x_outdatedOnGpu = true;
  xposdft_needsGpuEnsureCapacity = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInitStruct_cwtfilterbank(&fb, &x_emlrtRTEI, true);
  //  CWT scale spectrum - CUDA binary wrapper (double-precision, FP64)
  //  CWT uses morse wavelets, as they account for unequal variance-covariance
  //  across freqs. L1-norm is applied to mitigate 1/f decay of neuronal field
  //  potentials. Log-transform & robust z-scoring converts absolute valued CWT
  //  outputs into normal distribution.
  //
  //  Kevin Tan, 2022 (kevmtan.github.io)
  //  Input validation
  // 'ec_cwtAvg:11' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_cwtAvg:12' fs (1,1) double = 1000
  //  Sampling rate
  // 'ec_cwtAvg:13' fLims (1,2) double = [1 300]
  //  Frequency limits
  // 'ec_cwtAvg:14' fVoices (1,1) double = 10
  //  Voices per octave
  // 'ec_cwtAvg:15' ds (1,2) double = [1 1]
  //  Frequency limits
  // 'ec_cwtAvg:17' coder.gpu.kernelfun;
  // 'ec_cwtAvg:18' fVoices = round(fVoices);
  fVoices = std::round(fVoices);
  // 'ec_cwtAvg:19' nFrames = height(x);
  // 'ec_cwtAvg:20' nChs = width(x);
  // 'ec_cwtAvg:21' doDownsample = ds(2)>ds(1);
  doDownsample = (ds[1] > ds[0]);
  //  Prep CWT
  // 'ec_cwtAvg:24' fb = cwtfilterbank(Wavelet="Morse",SignalLength=nFrames,...
  // 'ec_cwtAvg:25'
  // SamplingFrequency=fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
  coder::cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(cpu_x->size[0]),
                                     fs, fLims, fVoices);
  // 'ec_cwtAvg:26' freqs = centerFrequencies(fb);
  i = freqs->size[0];
  freqs->size[0] = fb.WaveletCenterFrequencies->size[0];
  emxEnsureCapacity_real_T(freqs, i, &emlrtRTEI);
  for (i = 0; i < fb.WaveletCenterFrequencies->size[0]; i++) {
    freqs->data[i] = fb.WaveletCenterFrequencies->data[i];
  }
  //  Preallocate
  // 'ec_cwtAvg:29' tmp = cell(1,nChs);
  i = y->size[0] * y->size[1];
  y->size[0] = 1;
  y->size[1] = cpu_x->size[1];
  emxEnsureCapacity_cell_wrap_0(y, i, &b_emlrtRTEI);
  // 'ec_cwtAvg:30' y = coder.nullcopy(tmp);
  //  y = coder.nullcopy(x); % Preallocate output
  //  CWT average power (L1-norm & variance-norm usng weighted integrals)
  // 'ec_cwtAvg:34' for ch = 1:nChs
  i1 = cpu_x->size[1];
  emxInit_real32_T(&cpu_savgpTMP, 2, &y_emlrtRTEI, true);
  emxInit_real32_T(&cpu_Scales, 2, &c_emlrtRTEI, true);
  emxInit_real32_T(&cpu_psihat, 2, &d_emlrtRTEI, true);
  emxInit_real32_T(&cpu_xv, 2, &f_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_xposdft, 2, &g_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_cfsposdft, 2, &ab_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_x, 2, &j_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_cfs, 2, &l_emlrtRTEI, true);
  emxInit_real32_T(&cpu_y, 2, &bb_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_y, 2, &bb_emlrtRTEI, true);
  emxInit_real32_T(&cpu_abswt2S, 2, &p_emlrtRTEI, true);
  emxInit_real32_T(&c_cpu_y, 2, &q_emlrtRTEI, true);
  emxInit_real32_T(&cpu_c, 1, &t_emlrtRTEI, true);
  emxInit_real32_T(&cpu_xCh, 1, &u_emlrtRTEI, true);
  emxInit_real32_T(&cpu_r, 2, &cb_emlrtRTEI, true);
  emxInit_real32_T(&c_cpu_x, 1, &e_emlrtRTEI, true);
  emxInit_real32_T(&d_cpu_x, 1, &m_emlrtRTEI, true);
  for (int32_T ch{0}; ch < i1; ch++) {
    real_T cpsi;
    int32_T acoef;
    int32_T i3;
    int32_T i4;
    int32_T j2;
    int32_T loop_ub;
    int32_T nd2;
    real32_T c1;
    uint32_T xSize[2];
    uint32_T defaultSL_idx_1;
    boolean_T Scales_outdatedOnGpu;
    boolean_T psihat_outdatedOnGpu;
    boolean_T validLaunchParams;
    // 'ec_cwtAvg:35' xCh = scaleSpectrum(fb,x(:,ch),SpectrumType="density")';
    defaultSL_idx_1 = static_cast<uint32_T>(fb.Scales->size[1]);
    Scales_outdatedOnGpu = false;
    i = cpu_Scales->size[0] * cpu_Scales->size[1];
    cpu_Scales->size[0] = 1;
    cpu_Scales->size[1] = fb.Scales->size[1];
    emxEnsureCapacity_real32_T(cpu_Scales, i, &c_emlrtRTEI);
    for (i = 0; i < fb.Scales->size[1]; i++) {
      cpu_Scales->data[i] = static_cast<real32_T>(fb.Scales->data[i]);
      Scales_outdatedOnGpu = true;
    }
    psihat_outdatedOnGpu = false;
    i = cpu_psihat->size[0] * cpu_psihat->size[1];
    cpu_psihat->size[0] = fb.PsiDFT->size[0];
    cpu_psihat->size[1] = fb.PsiDFT->size[1];
    emxEnsureCapacity_real32_T(cpu_psihat, i, &d_emlrtRTEI);
    for (i = 0; i < fb.PsiDFT->size[0] * fb.PsiDFT->size[1]; i++) {
      cpu_psihat->data[i] = static_cast<real32_T>(fb.PsiDFT->data[i]);
      psihat_outdatedOnGpu = true;
    }
    nd2 = cpu_x->size[0];
    i = c_cpu_x->size[0];
    c_cpu_x->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real32_T(c_cpu_x, i, &e_emlrtRTEI);
    i = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    cpu_xv->size[1] = cpu_x->size[0];
    emxEnsureCapacity_real32_T(cpu_xv, i, &f_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nd2 - 1), &grid, &block,
                            2147483647U);
    if (x_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, true);
    r_outdatedOnCpu = false;
    gpuEmxEnsureCapacity_real32_T(c_cpu_x, &b_gpu_x, true);
    if (x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel1<<<grid, block>>>(ch, gpu_x, nd2 - 1, gpu_xv, b_gpu_x,
                                         cpu_x->size[0U]);
    }
    r_outdatedOnGpu = false;
    gpuEmxMemcpyGpuToCpu_real32_T(c_cpu_x, &b_gpu_x);
    fb.sigvar = coder::vvarstd(c_cpu_x, cpu_x->size[0]);
    if (fb.SignalPad > 0.0) {
      cpsi = fb.SignalPad;
      if (cpsi < 1.0) {
        acoef = 0;
      } else {
        acoef = static_cast<int32_T>(cpsi);
      }
      cpsi = (static_cast<real_T>(cpu_x->size[0]) - fb.SignalPad) + 1.0;
      if (cpsi > cpu_x->size[0]) {
        i3 = 0;
        i4 = 1;
        j2 = -1;
      } else {
        i3 = cpu_x->size[0] - 1;
        i4 = -1;
        j2 = static_cast<int32_T>(cpsi) - 1;
      }
      i = b_cpu_x->size[0] * b_cpu_x->size[1];
      b_cpu_x->size[0] = 1;
      b_cpu_x->size[1] = acoef;
      emxEnsureCapacity_real32_T(b_cpu_x, i, &j_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(acoef - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real32_T(b_cpu_x, &c_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel2<<<grid, block>>>(ch, gpu_x, acoef - 1, c_gpu_x,
                                           cpu_x->size[0U]);
      }
      mwGetLaunchParameters1D(computeNumIters((acoef >> 1) - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel3<<<grid, block>>>(acoef, (acoef >> 1) - 1, c_gpu_x);
      }
      i = cpu_xv->size[0] * cpu_xv->size[1];
      cpu_xv->size[0] = 1;
      cpu_xv->size[1] =
          ((cpu_x->size[0] + b_cpu_x->size[1]) + div_s32(j2 - i3, i4)) + 1;
      emxEnsureCapacity_real32_T(cpu_xv, i, &k_emlrtRTEI);
      loop_ub = b_cpu_x->size[1];
      mwGetLaunchParameters1D(computeNumIters(loop_ub - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel4<<<grid, block>>>(c_gpu_x, loop_ub - 1, gpu_xv);
      }
      xv_outdatedOnCpu = true;
      loop_ub = cpu_x->size[0];
      for (i = 0; i < loop_ub; i++) {
        if (xv_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_xv, &gpu_xv);
        }
        cpu_xv->data[i + b_cpu_x->size[1]] =
            cpu_x->data[i + cpu_x->size[0] * ch];
        xv_outdatedOnCpu = false;
        r_outdatedOnGpu = true;
        x_needsGpuEnsureCapacity = true;
        r_outdatedOnCpu = true;
      }
      loop_ub = div_s32(j2 - i3, i4);
      for (i = 0; i <= loop_ub; i++) {
        if (xv_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_xv, &gpu_xv);
        }
        cpu_xv->data[(i + cpu_x->size[0]) + b_cpu_x->size[1]] =
            cpu_x->data[(i3 + i4 * i) + cpu_x->size[0] * ch];
        xv_outdatedOnCpu = false;
        r_outdatedOnGpu = true;
        x_needsGpuEnsureCapacity = true;
        r_outdatedOnCpu = true;
      }
    }
    if (cpu_xv->size[1] == 0) {
      cpu_xposdft->size[0] = 1;
      cpu_xposdft->size[1] = 0;
    } else if (cpu_xv->size[1] -
                   (static_cast<int32_T>(
                        static_cast<uint32_T>(cpu_xv->size[1]) >> 1)
                    << 1) ==
               1) {
      hipfftHandle fftPlanHandle;
      i = cpu_xposdft->size[0] * cpu_xposdft->size[1];
      cpu_xposdft->size[0] = 1;
      cpu_xposdft->size[1] = cpu_xv->size[1];
      emxEnsureCapacity_creal32_T(cpu_xposdft, i, &g_emlrtRTEI);
      j2 = cpu_xv->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
      if (r_outdatedOnCpu) {
        gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, !r_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_creal32_T(cpu_xposdft, &gpu_xposdft, true);
      if (r_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_xv, cpu_xv);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel22<<<grid, block>>>(gpu_xv, j2, gpu_xposdft);
      }
      nd2 = cpu_xposdft->size[1];
      fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_C2C, 1);
      xposdft_needsGpuEnsureCapacity = false;
      hipfftExecC2C(fftPlanHandle, (hipfftComplex *)&gpu_xposdft.data[0],
                   (hipfftComplex *)&gpu_xposdft.data[0], HIPFFT_FORWARD);
    } else {
      hipfftHandle b_fftPlanHandle;
      nd2 = cpu_xv->size[1];
      for (j2 = 0; j2 < 2; j2++) {
        xSize[j2] = static_cast<uint32_T>(cpu_xv->size[j2]);
      }
      i = cpu_xposdft->size[0] * cpu_xposdft->size[1];
      cpu_xposdft->size[0] = 1;
      cpu_xposdft->size[1] = static_cast<int32_T>(xSize[1]);
      emxEnsureCapacity_creal32_T(cpu_xposdft, i, &h_emlrtRTEI);
      b_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_R2C, 1);
      if (r_outdatedOnCpu) {
        gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, !r_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_creal32_T(cpu_xposdft, &gpu_xposdft, true);
      if (r_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_xv, cpu_xv);
      }
      hipfftExecR2C(b_fftPlanHandle, (hipfftReal *)&gpu_xv.data[0],
                   (hipfftComplex *)&gpu_xposdft.data[0]);
      mwGetLaunchParameters1D(
          computeNumIters(
              static_cast<int32_T>(
                  static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) /
                  2.0) -
              2),
          &grid, &block, 2147483647U);
      xposdft_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel23<<<grid, block>>>(
            static_cast<int32_T>(
                static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) / 2.0) -
                2,
            gpu_xposdft, xSize[1]);
      }
    }
    nd2 = cpu_psihat->size[1];
    acoef = cpu_xposdft->size[1];
    if (nd2 <= acoef) {
      acoef = nd2;
    }
    if (cpu_psihat->size[1] == 1) {
      i = cpu_xposdft->size[1];
    } else if (cpu_xposdft->size[1] == 1) {
      i = cpu_psihat->size[1];
    } else if (cpu_xposdft->size[1] == cpu_psihat->size[1]) {
      i = cpu_xposdft->size[1];
    } else {
      i = acoef;
    }
    i4 = cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
    cpu_cfsposdft->size[0] = cpu_psihat->size[0];
    nd2 = cpu_psihat->size[1];
    acoef = cpu_xposdft->size[1];
    if (nd2 <= acoef) {
      acoef = nd2;
    }
    if (cpu_psihat->size[1] == 1) {
      cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
    } else if (cpu_xposdft->size[1] == 1) {
      cpu_cfsposdft->size[1] = cpu_psihat->size[1];
    } else if (cpu_xposdft->size[1] == cpu_psihat->size[1]) {
      cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
    } else {
      cpu_cfsposdft->size[1] = acoef;
    }
    emxEnsureCapacity_creal32_T(cpu_cfsposdft, i4, &i_emlrtRTEI);
    r_outdatedOnCpu = true;
    if ((cpu_psihat->size[0] != 0) && (i != 0)) {
      acoef = (cpu_xposdft->size[1] != 1);
      nd2 = (cpu_psihat->size[1] != 1);
      j2 = cpu_cfsposdft->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i - 1, j2), &grid, &block,
                              2147483647U);
      if (xposdft_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_creal32_T(cpu_xposdft, &gpu_xposdft, true);
      }
      xposdft_needsGpuEnsureCapacity = false;
      gpuEmxEnsureCapacity_real32_T(cpu_psihat, &gpu_psihat,
                                    !psihat_outdatedOnGpu);
      gpuEmxEnsureCapacity_creal32_T(cpu_cfsposdft, &gpu_cfsposdft, true);
      r_outdatedOnCpu = false;
      if (psihat_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_psihat, cpu_psihat);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel5<<<grid, block>>>(
            gpu_xposdft, gpu_psihat, nd2, acoef, j2, i - 1, gpu_cfsposdft,
            cpu_psihat->size[0U], cpu_cfsposdft->size[0U]);
      }
    }
    if ((cpu_cfsposdft->size[0] == 0) || (cpu_cfsposdft->size[1] == 0)) {
      for (j2 = 0; j2 < 2; j2++) {
        xSize[j2] = static_cast<uint32_T>(cpu_cfsposdft->size[j2]);
      }
      xSize[1] = static_cast<uint32_T>(cpu_cfsposdft->size[1]);
      i = cpu_cfs->size[0] * cpu_cfs->size[1];
      cpu_cfs->size[0] = static_cast<int32_T>(xSize[0]);
      cpu_cfs->size[1] = cpu_cfsposdft->size[1];
      emxEnsureCapacity_creal32_T(cpu_cfs, i, &l_emlrtRTEI);
      j2 = static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1;
      mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal32_T(cpu_cfs, &gpu_cfs, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel6<<<grid, block>>>(j2, gpu_cfs);
      }
      r_outdatedOnCpu = true;
    } else {
      hipfftHandle c_fftPlanHandle;
      nd2 = cpu_cfsposdft->size[1];
      c_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, cpu_cfsposdft->size[0],
                                         1, HIPFFT_C2C, cpu_cfsposdft->size[0]);
      if (r_outdatedOnCpu) {
        gpuEmxEnsureCapacity_creal32_T(cpu_cfsposdft, &gpu_cfsposdft, true);
      }
      hipfftExecC2C(c_fftPlanHandle, (hipfftComplex *)&gpu_cfsposdft.data[0],
                   (hipfftComplex *)&gpu_cfsposdft.data[0], HIPFFT_BACKWARD);
      i = cpu_cfs->size[0] * cpu_cfs->size[1];
      cpu_cfs->size[0] = cpu_cfsposdft->size[0];
      cpu_cfs->size[1] = cpu_cfsposdft->size[1];
      emxEnsureCapacity_creal32_T(cpu_cfs, i, &l_emlrtRTEI);
      c1 = static_cast<real32_T>(cpu_cfsposdft->size[1]);
      j2 = cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal32_T(cpu_cfs, &gpu_cfs, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel21<<<grid, block>>>(c1, gpu_cfsposdft, j2, gpu_cfs);
      }
      r_outdatedOnCpu = true;
    }
    if (fb.SignalPad > 0.0) {
      real_T d3;
      cpsi = fb.SignalPad + 1.0;
      d3 = fb.SignalPad + fb.SignalLength;
      if (cpsi > d3) {
        nd2 = 0;
        acoef = 0;
      } else {
        nd2 = static_cast<int32_T>(cpsi) - 1;
        acoef = static_cast<int32_T>(d3);
      }
      i3 = cpu_cfs->size[0];
      loop_ub = acoef - nd2;
      for (i = 0; i < loop_ub; i++) {
        for (i4 = 0; i4 < i3; i4++) {
          if (r_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_creal32_T(cpu_cfs, &gpu_cfs);
          }
          cpu_cfs->data[i4 + i3 * i] =
              cpu_cfs->data[i4 + cpu_cfs->size[0] * (nd2 + i)];
          r_outdatedOnCpu = false;
        }
      }
      i = cpu_cfs->size[0] * cpu_cfs->size[1];
      cpu_cfs->size[1] = acoef - nd2;
      emxEnsureCapacity_creal32_T(cpu_cfs, i, &l_emlrtRTEI);
    }
    cpsi =
        coder::wavelet::internal::cwt::numCpsi(fb.Wavelet, fb.Gamma, fb.Beta);
    i = d_cpu_x->size[0];
    d_cpu_x->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real32_T(d_cpu_x, i, &m_emlrtRTEI);
    j2 = cpu_x->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
    if (x_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, true);
    }
    x_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real32_T(d_cpu_x, &d_gpu_x, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel7<<<grid, block>>>(ch, gpu_x, j2, d_gpu_x,
                                         cpu_x->size[0U]);
    }
    if (r_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_creal32_T(cpu_cfs, &gpu_cfs);
    }
    gpuEmxMemcpyGpuToCpu_real32_T(d_cpu_x, &d_gpu_x);
    coder::wavelet::internal::cwt::scNormalize(
        cpu_cfs, cpsi, cpu_Scales, coder::vvarstd(d_cpu_x, cpu_x->size[0]));
    nd2 = cpu_cfs->size[0] * cpu_cfs->size[1];
    for (j2 = 0; j2 < 2; j2++) {
      xSize[j2] = static_cast<uint32_T>(cpu_cfs->size[j2]);
    }
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = static_cast<int32_T>(xSize[0]);
    cpu_y->size[1] = static_cast<int32_T>(xSize[1]);
    emxEnsureCapacity_real32_T(cpu_y, i, &n_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nd2 - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_cfs, &gpu_cfs, false);
    gpuEmxEnsureCapacity_real32_T(cpu_y, &gpu_y, true);
    gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_cfs, cpu_cfs);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel8<<<grid, block>>>(gpu_cfs, nd2 - 1, gpu_y);
    }
    for (j2 = 0; j2 < 2; j2++) {
      xSize[j2] = static_cast<uint32_T>(cpu_y->size[j2]);
    }
    i = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = static_cast<int32_T>(xSize[0]);
    b_cpu_y->size[1] = static_cast<int32_T>(xSize[1]);
    emxEnsureCapacity_real32_T(b_cpu_y, i, &o_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(xSize[0]) *
                                                static_cast<int32_T>(xSize[1]) -
                                            1),
                            &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(b_cpu_y, &b_gpu_y, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel9<<<grid, block>>>(
          gpu_y,
          static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1,
          b_gpu_y);
    }
    if (b_cpu_y->size[0] == cpu_Scales->size[1]) {
      i = cpu_abswt2S->size[0] * cpu_abswt2S->size[1];
      cpu_abswt2S->size[0] = b_cpu_y->size[0];
      cpu_abswt2S->size[1] = b_cpu_y->size[1];
      emxEnsureCapacity_real32_T(cpu_abswt2S, i, &p_emlrtRTEI);
      j2 = b_cpu_y->size[1] - 1;
      i3 = b_cpu_y->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(j2, i3), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_Scales, &gpu_Scales,
                                    !Scales_outdatedOnGpu);
      r_outdatedOnGpu = false;
      gpuEmxEnsureCapacity_real32_T(cpu_abswt2S, &gpu_abswt2S, true);
      xv_outdatedOnCpu = false;
      if (Scales_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_Scales, cpu_Scales);
      }
      Scales_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel10<<<grid, block>>>(gpu_Scales, b_gpu_y, i3, j2,
                                            gpu_abswt2S, cpu_abswt2S->size[0U],
                                            b_cpu_y->size[0U]);
      }
      r_outdatedOnCpu = false;
      abswt2S_outdatedOnCpu = true;
    } else {
      if (abswt2S_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_abswt2S, &gpu_abswt2S);
      }
      gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_y, &b_gpu_y);
      binary_expand_op(cpu_abswt2S, b_cpu_y, cpu_Scales);
      abswt2S_outdatedOnCpu = false;
      r_outdatedOnCpu = true;
      r_outdatedOnGpu = true;
      xv_outdatedOnCpu = true;
    }
    i = c_cpu_y->size[0] * c_cpu_y->size[1];
    c_cpu_y->size[0] = static_cast<int32_T>(defaultSL_idx_1);
    c_cpu_y->size[1] = cpu_abswt2S->size[1];
    emxEnsureCapacity_real32_T(c_cpu_y, i, &q_emlrtRTEI);
    j2 = cpu_abswt2S->size[1] - 1;
    mwGetLaunchParameters1D(
        computeNumIters(j2, static_cast<int32_T>(defaultSL_idx_1) - 1), &grid,
        &block, 2147483647U);
    if (xv_outdatedOnCpu) {
      gpuEmxEnsureCapacity_real32_T(cpu_abswt2S, &gpu_abswt2S,
                                    !r_outdatedOnCpu);
    }
    gpuEmxEnsureCapacity_real32_T(c_cpu_y, &c_gpu_y, true);
    if (r_outdatedOnCpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_abswt2S, cpu_abswt2S);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel11<<<grid, block>>>(
          gpu_abswt2S, static_cast<int32_T>(defaultSL_idx_1) - 1, j2, c_gpu_y,
          c_cpu_y->size[0U], cpu_abswt2S->size[0U]);
    }
    if ((static_cast<int32_T>(defaultSL_idx_1) == 0) &&
        (cpu_abswt2S->size[1] == 0)) {
      if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
        c1 = cpu_Scales->data[0] * 0.0F;
      } else {
        c1 = 0.0F;
      }
      i = cpu_savgpTMP->size[0] * cpu_savgpTMP->size[1];
      cpu_savgpTMP->size[0] = 1;
      cpu_savgpTMP->size[1] = 1;
      emxEnsureCapacity_real32_T(cpu_savgpTMP, i, &r_emlrtRTEI);
      gpuEmxEnsureCapacity_real32_T(cpu_savgpTMP, &gpu_savgpTMP, true);
      ec_cwtAvg_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(c1,
                                                                  gpu_savgpTMP);
    } else {
      i = cpu_savgpTMP->size[0] * cpu_savgpTMP->size[1];
      cpu_savgpTMP->size[0] = 1;
      cpu_savgpTMP->size[1] = cpu_abswt2S->size[1];
      emxEnsureCapacity_real32_T(cpu_savgpTMP, i, &r_emlrtRTEI);
      j2 = cpu_abswt2S->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_savgpTMP, &gpu_savgpTMP, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel14<<<grid, block>>>(j2, gpu_savgpTMP);
      }
      if (static_cast<int32_T>(defaultSL_idx_1) <= 1) {
        if ((static_cast<int32_T>(defaultSL_idx_1) == 1) &&
            (std::isinf(cpu_Scales->data[0]) ||
             std::isnan(cpu_Scales->data[0]))) {
          i = cpu_savgpTMP->size[0] * cpu_savgpTMP->size[1];
          cpu_savgpTMP->size[0] = 1;
          cpu_savgpTMP->size[1] = cpu_abswt2S->size[1];
          emxEnsureCapacity_real32_T(cpu_savgpTMP, i, &r_emlrtRTEI);
          j2 = cpu_abswt2S->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_savgpTMP, &gpu_savgpTMP, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_cwtAvg_kernel15<<<grid, block>>>(j2, gpu_savgpTMP);
          }
        }
      } else {
        if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
          i = cpu_c->size[0];
          cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
          emxEnsureCapacity_real32_T(cpu_c, i, &t_emlrtRTEI);
          mwGetLaunchParameters1D(
              computeNumIters(static_cast<int32_T>(defaultSL_idx_1) - 1), &grid,
              &block, 2147483647U);
          if (r_outdatedOnGpu) {
            gpuEmxEnsureCapacity_real32_T(cpu_Scales, &gpu_Scales,
                                          !Scales_outdatedOnGpu);
          }
          gpuEmxEnsureCapacity_real32_T(cpu_c, &gpu_c, true);
          if (Scales_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_Scales, cpu_Scales);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_cwtAvg_kernel16<<<grid, block>>>(
                gpu_Scales, static_cast<int32_T>(defaultSL_idx_1) - 1, gpu_c);
          }
          c1 = 0.5F * cpu_Scales->data[0];
          ec_cwtAvg_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              static_cast<int32_T>(defaultSL_idx_1), c1, gpu_c);
        } else {
          i = cpu_c->size[0];
          cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
          emxEnsureCapacity_real32_T(cpu_c, i, &s_emlrtRTEI);
          if (r_outdatedOnGpu) {
            gpuEmxEnsureCapacity_real32_T(cpu_Scales, &gpu_Scales,
                                          !Scales_outdatedOnGpu);
          }
          gpuEmxEnsureCapacity_real32_T(cpu_c, &gpu_c, true);
          if (Scales_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_Scales, cpu_Scales);
          }
          ec_cwtAvg_kernel18<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              gpu_Scales, gpu_c);
          mwGetLaunchParameters1D(
              computeNumIters(static_cast<int32_T>(defaultSL_idx_1) - 3), &grid,
              &block, 2147483647U);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_cwtAvg_kernel19<<<grid, block>>>(
                gpu_Scales, static_cast<int32_T>(defaultSL_idx_1) - 3, gpu_c);
          }
          ec_cwtAvg_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              gpu_Scales, static_cast<int32_T>(defaultSL_idx_1), gpu_c);
        }
        if (cpu_abswt2S->size[1] >= 1) {
          real32_T alpha1;
          c1 = 0.0F;
          alpha1 = 1.0F;
          cublasCheck(hipblasSgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                                  static_cast<int32_T>(defaultSL_idx_1),
                                  cpu_abswt2S->size[1], (float *)&alpha1,
                                  (float *)&c_gpu_y.data[0],
                                  static_cast<int32_T>(defaultSL_idx_1),
                                  (float *)&gpu_c.data[0], 1, (float *)&c1,
                                  (float *)&gpu_savgpTMP.data[0], 1),
                      __FILE__, __LINE__);
        }
      }
    }
    i = cpu_xCh->size[0];
    cpu_xCh->size[0] = cpu_savgpTMP->size[1];
    emxEnsureCapacity_real32_T(cpu_xCh, i, &u_emlrtRTEI);
    j2 = cpu_savgpTMP->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_xCh, &gpu_xCh, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel13<<<grid, block>>>(gpu_savgpTMP, j2, gpu_xCh);
    }
    xv_outdatedOnCpu = false;
    psihat_outdatedOnGpu = true;
    //  Downsample
    // 'ec_cwtAvg:38' if doDownsample
    if (doDownsample) {
      // 'ec_cwtAvg:39' y{ch} = resample(xCh,ds(1),ds(2));
      coder::resample(cpu_xCh, &gpu_xCh, &xv_outdatedOnCpu, ds[0], ds[1], cpu_r,
                      &r_outdatedOnCpu, &gpu_r, &r_outdatedOnGpu);
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = cpu_r->size[0];
      emxEnsureCapacity_real32_T(y->data[ch].f1, i, &w_emlrtRTEI);
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = 1;
      emxEnsureCapacity_real32_T(y->data[ch].f1, i, &w_emlrtRTEI);
      for (i = 0; i < cpu_r->size[0]; i++) {
        if (r_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_r, &gpu_r);
        }
        r_outdatedOnCpu = false;
        y->data[ch].f1->data[i] = cpu_r->data[i];
      }
      //  Downsample
    } else {
      // 'ec_cwtAvg:40' else
      // 'ec_cwtAvg:41' y{ch} = xCh;
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = cpu_xCh->size[0];
      emxEnsureCapacity_real32_T(y->data[ch].f1, i, &v_emlrtRTEI);
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = 1;
      emxEnsureCapacity_real32_T(y->data[ch].f1, i, &v_emlrtRTEI);
      for (i = 0; i < cpu_xCh->size[0]; i++) {
        if (psihat_outdatedOnGpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_xCh, &gpu_xCh);
        }
        psihat_outdatedOnGpu = false;
        y->data[ch].f1->data[i] = cpu_xCh->data[i];
      }
    }
  }
  emxFree_real32_T(&d_cpu_x);
  emxFree_real32_T(&c_cpu_x);
  emxFree_real32_T(&cpu_r);
  emxFree_real32_T(&cpu_xCh);
  emxFree_real32_T(&cpu_c);
  emxFree_real32_T(&c_cpu_y);
  emxFree_real32_T(&cpu_abswt2S);
  emxFree_real32_T(&b_cpu_y);
  emxFree_real32_T(&cpu_y);
  emxFree_creal32_T(&cpu_cfs);
  emxFree_real32_T(&b_cpu_x);
  emxFree_creal32_T(&cpu_cfsposdft);
  emxFree_creal32_T(&cpu_xposdft);
  emxFree_real32_T(&cpu_xv);
  emxFree_real32_T(&cpu_psihat);
  emxFree_real32_T(&cpu_Scales);
  emxFree_real32_T(&cpu_savgpTMP);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_x);
  gpuEmxFree_real32_T(&gpu_savgpTMP);
  gpuEmxFree_real32_T(&gpu_Scales);
  gpuEmxFree_real32_T(&gpu_psihat);
  gpuEmxFree_real32_T(&gpu_xv);
  gpuEmxFree_creal32_T(&gpu_xposdft);
  gpuEmxFree_creal32_T(&gpu_cfsposdft);
  gpuEmxFree_real32_T(&c_gpu_x);
  gpuEmxFree_creal32_T(&gpu_cfs);
  gpuEmxFree_real32_T(&gpu_y);
  gpuEmxFree_real32_T(&b_gpu_y);
  gpuEmxFree_real32_T(&gpu_abswt2S);
  gpuEmxFree_real32_T(&c_gpu_y);
  gpuEmxFree_real32_T(&gpu_c);
  gpuEmxFree_real32_T(&gpu_xCh);
  gpuEmxFree_real32_T(&gpu_r);
  gpuEmxFree_real32_T(&b_gpu_x);
  gpuEmxFree_real32_T(&d_gpu_x);
}

// End of code generation (ec_cwtAvg.cu)

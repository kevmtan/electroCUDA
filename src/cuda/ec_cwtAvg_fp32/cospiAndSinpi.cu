#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cospiAndSinpi.cu
//
// Code generation for function 'cospiAndSinpi'
//

// Include files
#include "cospiAndSinpi.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
namespace internal {
namespace scalar {
real_T cospiAndSinpi(real_T x, real_T *s)
{
  real_T c;
  boolean_T negateSinpi;
  if (x < 0.0) {
    x = -x;
    negateSinpi = true;
  } else {
    negateSinpi = false;
  }
  if (x < 0.25) {
    c = std::cos(x * 3.1415926535897931);
    *s = std::sin(x * 3.1415926535897931);
  } else {
    real_T r;
    r = x - 2.0 * std::floor(x / 2.0);
    if (r < 0.25) {
      r *= 3.1415926535897931;
      c = std::cos(r);
      *s = std::sin(r);
    } else if (r < 0.75) {
      r = 0.5 - r;
      r *= 3.1415926535897931;
      c = std::sin(r);
      *s = std::cos(r);
    } else if (r < 1.25) {
      r = 1.0 - r;
      r *= 3.1415926535897931;
      c = -std::cos(r);
      *s = std::sin(r);
    } else if (r < 1.75) {
      r -= 1.5;
      r *= 3.1415926535897931;
      c = std::sin(r);
      *s = -std::cos(r);
    } else {
      r -= 2.0;
      r *= 3.1415926535897931;
      c = std::cos(r);
      *s = std::sin(r);
    }
  }
  if (negateSinpi) {
    *s = -*s;
  }
  return c;
}

} // namespace scalar
} // namespace internal
} // namespace coder

// End of code generation (cospiAndSinpi.cu)

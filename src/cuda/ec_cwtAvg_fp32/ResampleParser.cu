#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ResampleParser.cu
//
// Code generation for function 'ResampleParser'
//

// Include files
#include "ResampleParser.h"
#include "casyi.h"
#include "cmlri.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "firls.h"
#include "gammaln.h"
#include "rt_nonfinite.h"
#include "warning.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo wf_emlrtRTEI{
    816,              // lineNo
    22,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo xf_emlrtRTEI{
    37,                                                    // lineNo
    20,                                                    // colNo
    "kaiser",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/kaiser.m" // pName
};

static emlrtRTEInfo yf_emlrtRTEI{
    816,              // lineNo
    63,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo ag_emlrtRTEI{
    816,              // lineNo
    17,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo bg_emlrtRTEI{
    745,              // lineNo
    21,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo cg_emlrtRTEI{
    777,              // lineNo
    13,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo dg_emlrtRTEI{
    778,              // lineNo
    15,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo eg_emlrtRTEI{
    778,              // lineNo
    27,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo fg_emlrtRTEI{
    778,              // lineNo
    13,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace b_signal {
namespace internal {
namespace resample {
void c_ResampleParser_createFilterAn(const real_T *obj_p, const real_T *obj_q,
                                     emxArray_real32_T *obj_filter,
                                     emxArray_real32_T *obj_filterWithPadding,
                                     real_T *obj_filterDelay,
                                     const emxArray_real32_T *obj_x)
{
  emxArray_int32_T *iv;
  emxArray_real_T *b_dv1;
  emxArray_real_T *c_dv;
  emxArray_real_T *h1;
  emxArray_real_T *r;
  emxArray_uint32_T *y;
  creal_T tmp;
  creal_T zd;
  real_T b_dv[4];
  real_T ak;
  real_T fc;
  real_T pqmax;
  int32_T inw;
  int32_T iseven;
  int32_T nw;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  pqmax = std::fmax(*obj_p, *obj_q);
  fc = 0.5 / pqmax;
  pqmax = 20.0 * pqmax + 1.0;
  b_dv[0] = 0.0;
  b_dv[1] = 2.0 * fc;
  b_dv[2] = 2.0 * fc;
  b_dv[3] = 1.0;
  emxInit_real_T(&r, 2, &wf_emlrtRTEI, true);
  firls(pqmax - 1.0, b_dv, r);
  emxInit_real_T(&c_dv, 1, &wf_emlrtRTEI, true);
  iseven = c_dv->size[0];
  c_dv->size[0] = r->size[1];
  emxEnsureCapacity_real_T(c_dv, iseven, &wf_emlrtRTEI);
  for (iseven = 0; iseven < r->size[1]; iseven++) {
    c_dv->data[iseven] = r->data[iseven];
  }
  emxFree_real_T(&r);
  if (pqmax == std::floor(pqmax)) {
    nw = static_cast<int32_T>(pqmax);
  } else {
    nw = static_cast<int32_T>(std::round(pqmax));
    ::coder::internal::warning();
  }
  emxInit_real_T(&b_dv1, 1, &wf_emlrtRTEI, true);
  iseven = b_dv1->size[0];
  b_dv1->size[0] = nw;
  emxEnsureCapacity_real_T(b_dv1, iseven, &xf_emlrtRTEI);
  if (nw <= 1) {
    iseven = b_dv1->size[0];
    b_dv1->size[0] = nw;
    emxEnsureCapacity_real_T(b_dv1, iseven, &yf_emlrtRTEI);
    for (iseven = 0; iseven < nw; iseven++) {
      b_dv1->data[iseven] = 1.0;
    }
  } else {
    int64_T b;
    int32_T mid;
    iseven = 1 - (nw & 1);
    mid = (nw >> 1) + 1;
    if (computeEndIdx(static_cast<int64_T>(mid), static_cast<int64_T>(nw),
                      1L) >= 0L) {
      zd.im = 0.0;
    }
    b = computeEndIdx(static_cast<int64_T>(mid), static_cast<int64_T>(nw), 1L);
    for (int64_T k{0L}; k <= b; k++) {
      pqmax = static_cast<real_T>(
                  iseven + ((static_cast<int32_T>(mid + k) - mid) << 1)) /
              (static_cast<real_T>(nw) - 1.0);
      zd.re = 5.0 * std::sqrt((1.0 - pqmax) * (pqmax + 1.0));
      if (std::isnan(zd.re)) {
        tmp.re = rtNaN;
        tmp.im = 0.0;
      } else {
        real_T az;
        int32_T b_nw;
        int32_T ierr;
        boolean_T guard1;
        ierr = 0;
        pqmax = rt_hypotd_snf(zd.re, 0.0);
        if (pqmax > 1.0737418235E+9) {
          ierr = 4;
        } else if (pqmax > 32767.999992370605) {
          ierr = 3;
        }
        tmp.re = 0.0;
        tmp.im = 0.0;
        az = rt_hypotd_snf(zd.re, 0.0);
        guard1 = false;
        if (az <= 2.0) {
          b_nw = 0;
          pqmax = rt_hypotd_snf(zd.re, 0.0);
          if (pqmax == 0.0) {
            tmp.re = 1.0;
            tmp.im = 0.0;
          } else if (pqmax < 2.2250738585072014E-305) {
            tmp.re = 1.0;
            tmp.im = 0.0;
          } else {
            real_T acz;
            real_T cz_re;
            fc = 0.5 * zd.re;
            if (pqmax > 4.7170688552396617E-153) {
              cz_re = fc * fc;
              acz = rt_hypotd_snf(cz_re, 0.0);
            } else {
              cz_re = 0.0;
              acz = 0.0;
            }
            pqmax = 1.0;
            gammaln(&pqmax);
            fc = std::log(fc) * 0.0 - pqmax;
            if (fc > -700.92179369444591) {
              real_T b_atol;
              real_T coef_im;
              real_T coef_re;
              real_T s1_im;
              real_T s1_re;
              coef_re = std::exp(fc);
              coef_im = std::exp(fc) * 0.0;
              b_atol = 2.2204460492503131E-16 * acz;
              s1_re = 1.0;
              s1_im = 0.0;
              if (!(acz < 2.2204460492503131E-16)) {
                real_T aa;
                real_T s;
                fc = 1.0;
                pqmax = 0.0;
                ak = 3.0;
                s = 1.0;
                aa = 2.0;
                real_T re;
                real_T rs;
                do {
                  rs = 1.0 / s;
                  re = fc * cz_re - pqmax * 0.0;
                  pqmax = fc * 0.0 + pqmax * cz_re;
                  fc = rs * re;
                  pqmax *= rs;
                  s1_re += fc;
                  s1_im += pqmax;
                  s += ak;
                  ak += 2.0;
                  aa = aa * acz * rs;
                } while (!!(aa > b_atol));
              }
              pqmax = s1_re * coef_re - s1_im * coef_im;
              s1_im = s1_re * coef_im + s1_im * coef_re;
              tmp.re = pqmax - s1_im * 0.0;
              tmp.im = pqmax * 0.0 + s1_im;
            } else {
              b_nw = 1;
              if (acz > 0.0) {
                b_nw = -1;
              }
            }
          }
          if (b_nw < 0) {
            inw = 1;
          } else {
            inw = b_nw;
          }
          if ((1 - inw != 0) && (b_nw < 0)) {
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1) {
          if (az < 21.784271729432426) {
            b_nw = cmlri(zd, 0.0, 1, &tmp);
            if (b_nw < 0) {
              if (b_nw == -2) {
                inw = -2;
              } else {
                inw = -1;
              }
            } else {
              inw = 0;
            }
          } else {
            b_nw = casyi(zd, 0.0, 1, &tmp);
            if (b_nw < 0) {
              if (b_nw == -2) {
                inw = -2;
              } else {
                inw = -1;
              }
            } else {
              inw = 0;
            }
          }
        }
        guard1 = false;
        if (inw < 0) {
          if (inw == -2) {
            tmp.re = rtNaN;
            tmp.im = 0.0;
          } else {
            ierr = 2;
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1 && (ierr == 2)) {
          tmp.re = rtInf;
          tmp.im = 0.0;
        }
        if (zd.re > 0.0) {
          pqmax = tmp.re;
          tmp.re = pqmax;
          tmp.im = 0.0;
        }
      }
      if (tmp.im == 0.0) {
        pqmax = tmp.re / 27.239871823604449;
        fc = 0.0;
      } else if (tmp.re == 0.0) {
        pqmax = 0.0;
        fc = tmp.im / 27.239871823604449;
      } else {
        pqmax = tmp.re / 27.239871823604449;
        fc = tmp.im / 27.239871823604449;
      }
      b_dv1->data[static_cast<int32_T>(mid + k) - 1] = rt_hypotd_snf(pqmax, fc);
    }
    for (iseven = 0; iseven <= mid - 2; iseven++) {
      b_dv1->data[iseven] = b_dv1->data[(nw - iseven) - 1];
    }
  }
  emxInit_real_T(&h1, 1, &ag_emlrtRTEI, true);
  iseven = h1->size[0];
  h1->size[0] = c_dv->size[0];
  emxEnsureCapacity_real_T(h1, iseven, &ag_emlrtRTEI);
  for (iseven = 0; iseven < c_dv->size[0]; iseven++) {
    h1->data[iseven] = c_dv->data[iseven] * b_dv1->data[iseven];
  }
  emxFree_real_T(&b_dv1);
  emxFree_real_T(&c_dv);
  inw = h1->size[0];
  if (h1->size[0] == 0) {
    pqmax = 0.0;
  } else {
    pqmax = h1->data[0];
    for (iseven = 0; iseven <= inw - 2; iseven++) {
      pqmax += h1->data[iseven + 1];
    }
  }
  iseven = obj_filter->size[0];
  obj_filter->size[0] = h1->size[0];
  emxEnsureCapacity_real32_T(obj_filter, iseven, &bg_emlrtRTEI);
  for (iseven = 0; iseven < h1->size[0]; iseven++) {
    obj_filter->data[iseven] =
        static_cast<real32_T>(*obj_p * h1->data[iseven] / pqmax);
  }
  emxFree_real_T(&h1);
  fc = (static_cast<real_T>(obj_filter->size[0]) - 1.0) / 2.0;
  pqmax = fc;
  if (*obj_q == 0.0) {
    if (fc == 0.0) {
      pqmax = 0.0;
    }
  } else if (std::isnan(*obj_q)) {
    pqmax = rtNaN;
  } else if (fc == 0.0) {
    pqmax = 0.0 / *obj_q;
  } else if (std::isinf(*obj_q)) {
    if (fc < 0.0) {
      pqmax = *obj_q;
    }
  } else {
    pqmax = std::fmod(fc, *obj_q);
    if (pqmax == 0.0) {
      pqmax = *obj_q * 0.0;
    } else if (fc < 0.0) {
      pqmax += *obj_q;
    }
  }
  ak = std::floor(*obj_q - pqmax);
  *obj_filterDelay = std::floor(std::ceil(fc + ak) / *obj_q);
  pqmax = static_cast<real_T>(obj_filter->size[0]) + ak;
  fc = 0.0;
  inw = obj_x->size[0];
  while (std::ceil((((static_cast<real_T>(inw) - 1.0) * *obj_p + pqmax) + fc) /
                   *obj_q) -
             *obj_filterDelay <
         std::ceil(static_cast<real_T>(inw) * *obj_p / *obj_q)) {
    fc++;
  }
  inw = static_cast<int32_T>((ak + static_cast<real_T>(obj_filter->size[0])) +
                             fc);
  iseven = obj_filterWithPadding->size[0];
  obj_filterWithPadding->size[0] = inw;
  emxEnsureCapacity_real32_T(obj_filterWithPadding, iseven, &cg_emlrtRTEI);
  if (inw - 1 >= 0) {
    std::memset(&obj_filterWithPadding->data[0], 0,
                static_cast<uint32_T>(inw) * sizeof(real32_T));
  }
  emxInit_uint32_T(&y, 2, &eg_emlrtRTEI, true);
  if (obj_filter->size[0] < 1) {
    y->size[0] = 1;
    y->size[1] = 0;
  } else {
    uint32_T u;
    u = static_cast<uint32_T>(obj_filter->size[0]);
    iseven = y->size[0] * y->size[1];
    y->size[0] = 1;
    y->size[1] = static_cast<int32_T>(u);
    emxEnsureCapacity_uint32_T(y, iseven, &gb_emlrtRTEI);
    for (iseven = 0; iseven < static_cast<int32_T>(u); iseven++) {
      y->data[iseven] = static_cast<uint32_T>(iseven) + 1U;
    }
  }
  emxInit_int32_T(&iv, 2, &fg_emlrtRTEI, true);
  iseven = iv->size[0] * iv->size[1];
  iv->size[0] = 1;
  iv->size[1] = y->size[1];
  emxEnsureCapacity_int32_T(iv, iseven, &dg_emlrtRTEI);
  for (iseven = 0; iseven < y->size[1]; iseven++) {
    iv->data[iseven] =
        static_cast<int32_T>(ak + static_cast<real_T>(y->data[iseven]));
  }
  emxFree_uint32_T(&y);
  for (iseven = 0; iseven < iv->size[1]; iseven++) {
    obj_filterWithPadding->data[iv->data[iseven] - 1] =
        obj_filter->data[iseven];
  }
  emxFree_int32_T(&iv);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

} // namespace resample
} // namespace internal
} // namespace b_signal
} // namespace coder

// End of code generation (ResampleParser.cu)

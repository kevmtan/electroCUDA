#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mldivide.cu
//
// Code generation for function 'mldivide'
//

// Include files
#include "mldivide.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "lapacke.h"
#include "hip/hip_math_constants.h"
#include <cmath>
#include <cstddef>

// Variable Definitions
static emlrtRTEInfo af_emlrtRTEI{
    20,                                                              // lineNo
    5,                                                               // colNo
    "mldivide",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/mldivide.m" // pName
};

static emlrtRTEInfo
    bf_emlrtRTEI{
        1,        // lineNo
        32,       // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo cf_emlrtRTEI{
    1,             // lineNo
    42,            // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

static emlrtRTEInfo
    df_emlrtRTEI{
        61,       // lineNo
        9,        // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo ef_emlrtRTEI{
    27,            // lineNo
    23,            // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

static emlrtRTEInfo
    ff_emlrtRTEI{
        92,       // lineNo
        22,       // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo
    gf_emlrtRTEI{
        105,      // lineNo
        1,        // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo hf_emlrtRTEI{
    68,            // lineNo
    9,             // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

static emlrtRTEInfo if_emlrtRTEI{
    85,        // lineNo
    26,        // colNo
    "qrsolve", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/qrsolve.m" // pName
};

static emlrtRTEInfo jf_emlrtRTEI{
    119,       // lineNo
    5,         // colNo
    "qrsolve", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/qrsolve.m" // pName
};

static emlrtRTEInfo kf_emlrtRTEI{
    1,                                                               // lineNo
    14,                                                              // colNo
    "mldivide",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/mldivide.m" // pName
};

static emlrtRTEInfo lf_emlrtRTEI{
    27,            // lineNo
    1,             // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

// Function Declarations
static void cusolverCheck(hipsolverStatus_t errCode, const char_T *file,
                          uint32_T b_line);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static __global__ void mldivide_kernel38(const int32_T b, const int32_T na,
                                         emxArray_real_T A, int32_T A_dim0);

static __global__ void mldivide_kernel39(const emxArray_real_T B,
                                         const emxArray_int32_T jpvt,
                                         const int32_T na, emxArray_real_T Y);

// Function Definitions
static void cusolverCheck(hipsolverStatus_t errCode, const char_T *file,
                          uint32_T b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPSOLVER_STATUS_SUCCESS) {
    cusolverGetErrorName(errCode, &errName);
    cusolverGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
}

static __global__ __launch_bounds__(1024, 1) void mldivide_kernel38(
    const int32_T b, const int32_T na, emxArray_real_T A, int32_T A_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(na) + 1UL) * (static_cast<uint64_T>(b) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b) + 1UL));
    A.data[j * A_dim0 + i] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void mldivide_kernel39(
    const emxArray_real_T B, const emxArray_int32_T jpvt, const int32_T na,
    emxArray_real_T Y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(na);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    Y.data[jpvt.data[i] - 1] = B.data[i];
  }
}

//
//
namespace coder {
void mldivide(const emxArray_real_T *A, const emxArray_real_T *B,
              emxArray_real_T *cpu_Y, boolean_T *Y_outdatedOnCpu,
              emxArray_real_T *gpu_Y, boolean_T *Y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_IPIV;
  emxArray_int32_T gpu_jpvt;
  emxArray_int32_T *cpu_IPIV;
  emxArray_int32_T *cpu_jpvt;
  emxArray_ptrdiff_t *jpvt_t;
  emxArray_real_T b_gpu_A;
  emxArray_real_T gpu_A;
  emxArray_real_T gpu_B;
  emxArray_real_T gpu_tau;
  emxArray_real_T *b_cpu_A;
  emxArray_real_T *cpu_A;
  emxArray_real_T *cpu_B;
  emxArray_real_T *cpu_tau;
  int32_T cpu_maxmn;
  int32_T *gpu_maxmn;
  checkCudaError(mwCudaMalloc(&gpu_maxmn, 4UL), __FILE__, __LINE__);
  gpuEmxReset_real_T(&gpu_B);
  gpuEmxReset_int32_T(&gpu_IPIV);
  gpuEmxReset_real_T(&gpu_A);
  gpuEmxReset_real_T(&gpu_tau);
  gpuEmxReset_int32_T(&gpu_jpvt);
  gpuEmxReset_real_T(&b_gpu_A);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&cpu_A, 2, &kf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_jpvt, 2, &kf_emlrtRTEI, true);
  emxInit_real_T(&cpu_tau, 1, &kf_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_A, 2, &kf_emlrtRTEI, true);
  emxInit_ptrdiff_t(&jpvt_t, 1, &gf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_IPIV, 1, &lf_emlrtRTEI, true);
  emxInit_real_T(&cpu_B, 1, &if_emlrtRTEI, true);
  if ((A->size[0] == 0) || (A->size[1] == 0) || (B->size[0] == 0)) {
    int32_T u0;
    *Y_outdatedOnCpu = false;
    *Y_outdatedOnGpu = false;
    u0 = cpu_Y->size[0];
    cpu_Y->size[0] = A->size[1];
    emxEnsureCapacity_real_T(cpu_Y, u0, &af_emlrtRTEI);
    for (u0 = 0; u0 < A->size[1]; u0++) {
      cpu_Y->data[u0] = 0.0;
      *Y_outdatedOnGpu = true;
    }
  } else if (A->size[0] == A->size[1]) {
    int32_T minmn;
    int32_T u0;
    boolean_T A_outdatedOnGpu;
    *Y_outdatedOnCpu = false;
    *Y_outdatedOnGpu = false;
    u0 = cpu_Y->size[0];
    cpu_Y->size[0] = B->size[0];
    emxEnsureCapacity_real_T(cpu_Y, u0, &af_emlrtRTEI);
    for (u0 = 0; u0 < B->size[0]; u0++) {
      cpu_Y->data[u0] = B->data[u0];
      *Y_outdatedOnGpu = true;
    }
    u0 = A->size[0];
    minmn = A->size[1];
    if (u0 <= minmn) {
      minmn = u0;
    }
    u0 = B->size[0];
    if (u0 <= minmn) {
      minmn = u0;
    }
    A_outdatedOnGpu = false;
    u0 = b_cpu_A->size[0] * b_cpu_A->size[1];
    b_cpu_A->size[0] = A->size[0];
    b_cpu_A->size[1] = A->size[1];
    emxEnsureCapacity_real_T(b_cpu_A, u0, &cf_emlrtRTEI);
    for (u0 = 0; u0 < A->size[0] * A->size[1]; u0++) {
      b_cpu_A->data[u0] = A->data[u0];
      A_outdatedOnGpu = true;
    }
    u0 = cpu_IPIV->size[0];
    cpu_IPIV->size[0] = minmn;
    emxEnsureCapacity_int32_T(cpu_IPIV, u0, &ef_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(b_cpu_A, &gpu_A, !A_outdatedOnGpu);
    if (A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_A, b_cpu_A);
    }
    cusolverCheck(hipsolverDnDgetrf_bufferSize(getCuSolverGlobalHandle(), minmn,
                                              minmn, (double *)&gpu_A.data[0],
                                              b_cpu_A->size[0],
                                              getCuSolverWorkspaceReq()),
                  __FILE__, __LINE__);
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    gpuEmxEnsureCapacity_int32_T(cpu_IPIV, &gpu_IPIV, true);
    cusolverCheck(
        hipsolverDnDgetrf(getCuSolverGlobalHandle(), minmn, minmn,
                         (double *)&gpu_A.data[0], b_cpu_A->size[0],
                         static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                         &gpu_IPIV.data[0], gpu_maxmn),
        __FILE__, __LINE__);
    checkCudaError(
        hipMemcpy(&cpu_maxmn, gpu_maxmn, 4UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    if (cpu_maxmn < 0) {
      *Y_outdatedOnGpu = false;
      u0 = cpu_Y->size[0];
      cpu_Y->size[0] = B->size[0];
      emxEnsureCapacity_real_T(cpu_Y, u0, &hf_emlrtRTEI);
      for (u0 = 0; u0 < B->size[0]; u0++) {
        cpu_Y->data[u0] = rtNaN;
        *Y_outdatedOnGpu = true;
      }
    } else {
      gpuEmxEnsureCapacity_real_T(cpu_Y, gpu_Y, !*Y_outdatedOnGpu);
      if (*Y_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(gpu_Y, cpu_Y);
      }
      cusolverCheck(hipsolverDnDgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N,
                                     minmn, 1, (double *)&gpu_A.data[0],
                                     b_cpu_A->size[0], &gpu_IPIV.data[0],
                                     (double *)&gpu_Y->data[0], B->size[0],
                                     gpu_maxmn),
                    __FILE__, __LINE__);
      *Y_outdatedOnGpu = false;
      *Y_outdatedOnCpu = true;
    }
  } else {
    ptrdiff_t info_t;
    real_T tol;
    int32_T minmana;
    int32_T minmn;
    int32_T na;
    int32_T u0;
    boolean_T A_outdatedOnCpu;
    boolean_T A_outdatedOnGpu;
    boolean_T B_outdatedOnGpu;
    boolean_T jpvt_outdatedOnGpu;
    boolean_T validLaunchParams;
    A_outdatedOnCpu = false;
    u0 = cpu_A->size[0] * cpu_A->size[1];
    cpu_A->size[0] = A->size[0];
    cpu_A->size[1] = A->size[1];
    emxEnsureCapacity_real_T(cpu_A, u0, &bf_emlrtRTEI);
    for (u0 = 0; u0 < A->size[0] * A->size[1]; u0++) {
      cpu_A->data[u0] = A->data[u0];
    }
    cpu_maxmn = cpu_A->size[0];
    na = cpu_A->size[1] - 1;
    jpvt_outdatedOnGpu = false;
    u0 = cpu_jpvt->size[0] * cpu_jpvt->size[1];
    cpu_jpvt->size[0] = 1;
    cpu_jpvt->size[1] = cpu_A->size[1];
    emxEnsureCapacity_int32_T(cpu_jpvt, u0, &df_emlrtRTEI);
    u0 = cpu_A->size[0];
    minmana = cpu_A->size[1];
    if (u0 <= minmana) {
      minmana = u0;
    }
    u0 = cpu_tau->size[0];
    cpu_tau->size[0] = minmana;
    emxEnsureCapacity_real_T(cpu_tau, u0, &ff_emlrtRTEI);
    u0 = jpvt_t->size[0];
    jpvt_t->size[0] = cpu_A->size[1];
    emxEnsureCapacity_ptrdiff_t(jpvt_t, u0, &gf_emlrtRTEI);
    for (u0 = 0; u0 < cpu_A->size[1]; u0++) {
      cpu_jpvt->data[u0] = 0;
      jpvt_outdatedOnGpu = true;
      jpvt_t->data[u0] = (ptrdiff_t)0;
    }
    info_t = LAPACKE_dgeqp3(102, (ptrdiff_t)cpu_A->size[0],
                            (ptrdiff_t)cpu_A->size[1], &cpu_A->data[0],
                            (ptrdiff_t)cpu_A->size[0], &jpvt_t->data[0],
                            &cpu_tau->data[0]);
    A_outdatedOnGpu = true;
    if ((int32_T)info_t != 0) {
      int64_T b;
      mwGetLaunchParameters1D(computeNumIters(na, cpu_maxmn - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_A, &b_gpu_A, false);
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_A, cpu_A);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        mldivide_kernel38<<<grid, block>>>(cpu_maxmn - 1, na, b_gpu_A,
                                           cpu_A->size[0U]);
      }
      A_outdatedOnGpu = false;
      A_outdatedOnCpu = true;
      u0 = cpu_A->size[0];
      minmn = cpu_A->size[1];
      if (u0 <= minmn) {
        minmn = u0;
      }
      for (cpu_maxmn = 0; cpu_maxmn < minmn; cpu_maxmn++) {
        cpu_tau->data[cpu_maxmn] = rtNaN;
      }
      b = computeEndIdx(static_cast<int64_T>(minmn + 1),
                        static_cast<int64_T>(minmana), 1L);
      for (int64_T k{0L}; k <= b; k++) {
        cpu_tau->data[static_cast<int32_T>((minmn + 1) + k) - 1] = 0.0;
      }
      for (cpu_maxmn = 0; cpu_maxmn <= na; cpu_maxmn++) {
        cpu_jpvt->data[cpu_maxmn] = cpu_maxmn + 1;
        jpvt_outdatedOnGpu = true;
      }
    } else {
      for (cpu_maxmn = 0; cpu_maxmn <= na; cpu_maxmn++) {
        cpu_jpvt->data[cpu_maxmn] = (int32_T)jpvt_t->data[cpu_maxmn];
        jpvt_outdatedOnGpu = true;
      }
    }
    na = -1;
    if (cpu_A->size[0] < cpu_A->size[1]) {
      minmn = cpu_A->size[0];
      cpu_maxmn = cpu_A->size[1];
    } else {
      minmn = cpu_A->size[1];
      cpu_maxmn = cpu_A->size[0];
    }
    if (A_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real_T(cpu_A, &b_gpu_A);
    }
    tol = std::fmin(1.4901161193847656E-8,
                    2.2204460492503131E-15 * static_cast<real_T>(cpu_maxmn)) *
          std::abs(cpu_A->data[0]);
    while ((na + 1 < minmn) &&
           (!(std::abs(cpu_A->data[(na + cpu_A->size[0] * (na + 1)) + 1]) <=
              tol))) {
      na++;
    }
    B_outdatedOnGpu = false;
    u0 = cpu_B->size[0];
    cpu_B->size[0] = B->size[0];
    emxEnsureCapacity_real_T(cpu_B, u0, &if_emlrtRTEI);
    for (u0 = 0; u0 < B->size[0]; u0++) {
      cpu_B->data[u0] = B->data[u0];
      B_outdatedOnGpu = true;
    }
    *Y_outdatedOnGpu = false;
    u0 = cpu_Y->size[0];
    cpu_Y->size[0] = cpu_A->size[1];
    emxEnsureCapacity_real_T(cpu_Y, u0, &af_emlrtRTEI);
    for (u0 = 0; u0 < cpu_A->size[1]; u0++) {
      cpu_Y->data[u0] = 0.0;
      *Y_outdatedOnGpu = true;
    }
    cpu_maxmn = 0;
    u0 = cpu_A->size[0];
    minmn = cpu_A->size[1];
    if (u0 <= minmn) {
      minmn = u0;
    }
    gpuEmxEnsureCapacity_real_T(cpu_A, &b_gpu_A, !A_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_tau, &gpu_tau, false);
    gpuEmxEnsureCapacity_real_T(cpu_B, &gpu_B, !B_outdatedOnGpu);
    if (A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_A, cpu_A);
    }
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_tau, cpu_tau);
    if (B_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_B, cpu_B);
    }
    cusolverCheck(hipsolverDnDormqr_bufferSize(
                      getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                      cpu_B->size[0], 1, minmn, (double *)&b_gpu_A.data[0],
                      cpu_A->size[0], (double *)&gpu_tau.data[0],
                      (double *)&gpu_B.data[0], cpu_B->size[0],
                      getCuSolverWorkspaceReq()),
                  __FILE__, __LINE__);
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    A_outdatedOnCpu = false;
    checkCudaError(
        hipMemcpy(gpu_maxmn, &cpu_maxmn, 4UL, hipMemcpyHostToDevice),
        __FILE__, __LINE__);
    cusolverCheck(hipsolverDnDormqr(
                      getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                      cpu_B->size[0], 1, minmn, (double *)&b_gpu_A.data[0],
                      cpu_A->size[0], (double *)&gpu_tau.data[0],
                      (double *)&gpu_B.data[0], cpu_B->size[0],
                      static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                      *getCuSolverWorkspaceReq(), gpu_maxmn),
                  __FILE__, __LINE__);
    B_outdatedOnGpu = false;
    checkCudaError(
        hipMemcpy(&cpu_maxmn, gpu_maxmn, 4UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    if (cpu_maxmn < 0) {
      cpu_maxmn = cpu_B->size[0];
      u0 = cpu_B->size[0];
      cpu_B->size[0] = cpu_maxmn;
      emxEnsureCapacity_real_T(cpu_B, u0, &jf_emlrtRTEI);
      A_outdatedOnCpu = true;
      for (u0 = 0; u0 < cpu_maxmn; u0++) {
        cpu_B->data[u0] = rtNaN;
        B_outdatedOnGpu = true;
      }
    }
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (A_outdatedOnCpu) {
      gpuEmxEnsureCapacity_real_T(cpu_B, &gpu_B, !B_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_int32_T(cpu_jpvt, &gpu_jpvt, !jpvt_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_Y, gpu_Y, !*Y_outdatedOnGpu);
    if (B_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_B, cpu_B);
    }
    if (jpvt_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_jpvt, cpu_jpvt);
    }
    if (*Y_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_Y, cpu_Y);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      mldivide_kernel39<<<grid, block>>>(gpu_B, gpu_jpvt, na, *gpu_Y);
    }
    *Y_outdatedOnGpu = false;
    *Y_outdatedOnCpu = true;
    for (minmn = 0; minmn <= na; minmn++) {
      u0 = cpu_jpvt->data[na - minmn];
      if (*Y_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_Y, gpu_Y);
      }
      cpu_Y->data[u0 - 1] /=
          cpu_A->data[(na - minmn) + cpu_A->size[0] * (na - minmn)];
      *Y_outdatedOnCpu = false;
      *Y_outdatedOnGpu = true;
      cpu_maxmn = na - minmn;
      for (u0 = 0; u0 < cpu_maxmn; u0++) {
        cpu_Y->data[cpu_jpvt->data[u0] - 1] -=
            cpu_Y->data[cpu_jpvt->data[na - minmn] - 1] *
            cpu_A->data[u0 + cpu_A->size[0] * (na - minmn)];
      }
    }
  }
  emxFree_real_T(&cpu_B);
  emxFree_int32_T(&cpu_IPIV);
  emxFree_ptrdiff_t(&jpvt_t);
  emxFree_real_T(&b_cpu_A);
  emxFree_real_T(&cpu_tau);
  emxFree_int32_T(&cpu_jpvt);
  emxFree_real_T(&cpu_A);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&b_gpu_A);
  gpuEmxFree_int32_T(&gpu_jpvt);
  gpuEmxFree_real_T(&gpu_tau);
  gpuEmxFree_real_T(&gpu_A);
  gpuEmxFree_int32_T(&gpu_IPIV);
  gpuEmxFree_real_T(&gpu_B);
  checkCudaError(mwCudaFree(gpu_maxmn), __FILE__, __LINE__);
}

} // namespace coder

// End of code generation (mldivide.cu)

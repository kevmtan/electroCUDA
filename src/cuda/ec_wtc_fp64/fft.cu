#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// fft.cu
//
// Code generation for function 'fft'
//

// Include files
#include "fft.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hipfft/hipfft.h"
#include "nvtx3/nvToolsExt.h"

// Variable Definitions
static emlrtRTEInfo td_emlrtRTEI{
    63,                                                             // lineNo
    5,                                                              // colNo
    "fft",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/fft.m" // pName
};

// Function Declarations
static __global__ void fft_kernel27(const uint32_T xSize_dim1, const int32_T b,
                                    emxArray_creal_T y);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void fft_kernel27(
    const uint32_T xSize_dim1, const int32_T b, emxArray_creal_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    y.data[(static_cast<int32_T>(xSize_dim1) - i) - 1].re = y.data[i + 1].re;
    y.data[(static_cast<int32_T>(xSize_dim1) - i) - 1].im = -y.data[i + 1].im;
  }
}

//
//
namespace coder {
void fft(emxArray_real_T *cpu_x, boolean_T *x_outdatedOnCpu,
         emxArray_real_T *gpu_x, boolean_T *x_outdatedOnGpu,
         emxArray_creal_T *cpu_y, boolean_T *y_outdatedOnCpu,
         emxArray_creal_T *gpu_y, boolean_T *y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  nvtxRangePushA("#fcn#fft#" MW_AT_LOCATION);
  if (cpu_x->size[1] == 0) {
    *y_outdatedOnCpu = false;
    cpu_y->size[0] = 1;
    cpu_y->size[1] = 0;
  } else if (cpu_x->size[1] - (static_cast<int32_T>(
                                   static_cast<uint32_T>(cpu_x->size[1]) >> 1)
                               << 1) ==
             1) {
    hipfftHandle b_fftPlanHandle;
    int32_T i;
    *y_outdatedOnGpu = false;
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_y, i, &td_emlrtRTEI);
    profileLoopStart("fft_loop_0", __LINE__, (cpu_x->size[1] - 1) + 1, "");
    for (i = 0; i < cpu_x->size[1]; i++) {
      if (*x_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_x, gpu_x);
      }
      *x_outdatedOnCpu = false;
      cpu_y->data[i].re = cpu_x->data[i];
      cpu_y->data[i].im = 0.0;
      *y_outdatedOnGpu = true;
    }
    int32_T midpoint;
    profileLoopEnd();
    midpoint = cpu_y->size[1];
    b_fftPlanHandle =
        acquireCUFFTPlan(1, &midpoint, &midpoint, 1, 1, HIPFFT_Z2Z, 1);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_y, gpu_y, !*y_outdatedOnGpu);
    if (*y_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_y, cpu_y);
    }
    hipfftExecZ2Z(b_fftPlanHandle, (hipfftDoubleComplex *)&gpu_y->data[0],
                 (hipfftDoubleComplex *)&gpu_y->data[0], HIPFFT_FORWARD);
    *y_outdatedOnCpu = true;
  } else {
    hipfftHandle fftPlanHandle;
    int32_T i;
    int32_T midpoint;
    uint32_T xSize[2];
    boolean_T validLaunchParams;
    midpoint = cpu_x->size[1];
    xSize[1] = static_cast<uint32_T>(cpu_x->size[1]);
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_y, i, &ud_emlrtRTEI);
    fftPlanHandle =
        acquireCUFFTPlan(1, &midpoint, &midpoint, 1, 1, HIPFFT_D2Z, 1);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_y, gpu_y, true);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_x, cpu_x);
    }
    *x_outdatedOnGpu = false;
    hipfftExecD2Z(fftPlanHandle, (hipfftDoubleReal *)&gpu_x->data[0],
                 (hipfftDoubleComplex *)&gpu_y->data[0]);
    midpoint =
        static_cast<int32_T>(static_cast<real_T>(cpu_x->size[1] + 1) / 2.0);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(midpoint - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#fft_kernel27#" MW_AT_LINE);
      fft_kernel27<<<grid, block>>>(xSize[1], midpoint - 2, *gpu_y);
    }
    *y_outdatedOnCpu = true;
  }
  *y_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace coder

// End of code generation (fft.cu)

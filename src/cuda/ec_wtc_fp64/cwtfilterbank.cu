#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtfilterbank.cu
//
// Code generation for function 'cwtfilterbank'
//

// Include files
#include "cwtfilterbank.h"
#include "cwtfreqlimits.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "log2.h"
#include "rt_nonfinite.h"
#include "wavCFandSD.h"
#include "wavbpfilters.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtMCInfo b_emlrtMCI{
    53,                                                                // lineNo
    14,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtMCInfo c_emlrtMCI{
    55,                                                                // lineNo
    15,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtRTEInfo wb_emlrtRTEI{
    999,             // lineNo
    17,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo xb_emlrtRTEI{
    1000,            // lineNo
    17,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo yb_emlrtRTEI{
    1009,            // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo ac_emlrtRTEI{
    1278,            // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo bc_emlrtRTEI{
    952,             // lineNo
    34,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo cc_emlrtRTEI{
    952,             // lineNo
    45,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo dc_emlrtRTEI{
    217,             // lineNo
    21,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo ec_emlrtRTEI{
    18,             // lineNo
    5,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo gc_emlrtRTEI{
    31,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo hc_emlrtRTEI{
    26,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo ic_emlrtRTEI{
    33,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo jc_emlrtRTEI{
    27,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo kc_emlrtRTEI{
    956,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo lc_emlrtRTEI{
    227,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo nc_emlrtRTEI{
    228,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo oc_emlrtRTEI{
    1276,            // lineNo
    30,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo pc_emlrtRTEI{
    1276,            // lineNo
    25,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo qc_emlrtRTEI{
    16,             // lineNo
    5,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    32,             // lineNo
    18,             // colNo
    "getCWTScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "getCWTScales.m" // pName
};

static emlrtRTEInfo sc_emlrtRTEI{
    32,             // lineNo
    13,             // colNo
    "getCWTScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "getCWTScales.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    34,             // lineNo
    42,             // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    934,             // lineNo
    26,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    816,             // lineNo
    25,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret);

namespace coder {
static cwtfilterbank *cwtfilterbank_setProperties(cwtfilterbank *self,
                                                  real_T varargin_2,
                                                  const real_T varargin_6[2],
                                                  real_T varargin_8,
                                                  real_T varargin_10);

}
static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y);

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location);

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret)
{
  static const int32_T dims[2]{1, -1};
  int32_T iv[2];
  int32_T i;
  boolean_T bv[2]{false, true};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                            (const void *)&dims[0], &bv[0], &iv[0]);
  i = ret->size[0] * ret->size[1];
  ret->size[0] = iv[0];
  ret->size[1] = iv[1];
  nvtxMarkA("#emxEnsureCapacity_char_T#" MW_AT_LINE);
  emxEnsureCapacity_char_T(ret, i, static_cast<emlrtRTEInfo *>(nullptr));
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret->data[0], 1, false);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

//
//
namespace coder {
static cwtfilterbank *cwtfilterbank_setProperties(cwtfilterbank *self,
                                                  real_T varargin_2,
                                                  const real_T varargin_6[2],
                                                  real_T varargin_8,
                                                  real_T varargin_10)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 2};
  static const int32_T iv2[2]{1, 7};
  static const int32_T iv3[2]{1, 5};
  static const char_T b[10]{'r', 'e', 'f', 'l', 'e', 'c', 't', 'i', 'o', 'n'};
  static const char_T cv2[10]{'r', 'e', 'f', 'l', 'e', 'c', 't', 'i', 'o', 'n'};
  static const char_T u[7]{'s', 'p', 'r', 'i', 'n', 't', 'f'};
  static const char_T b_formatSpec[5]{'%', '2', '.', '2', 'f'};
  static const char_T cv3[4]{'b', 'u', 'm', 'p'};
  static const char_T cv4[4]{'a', 'm', 'o', 'r'};
  static const char_T formatSpec[2]{'%', 'f'};
  cwtfilterbank *b_self;
  emxArray_char_T *d_tmpStr;
  emxArray_char_T *tmpStr;
  const mxArray *b_tmpStr;
  const mxArray *b_y;
  const mxArray *c_tmpStr;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *f_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *m4;
  const mxArray *m5;
  const mxArray *y;
  real_T NyquistRange_idx_1;
  real_T cf;
  real_T sigmat;
  int32_T exitg2;
  int32_T k;
  char_T a[10];
  boolean_T b_b[2];
  boolean_T exitg1;
  boolean_T freqsep;
  boolean_T guard1;
  nvtxRangePushA("#fcn#cwtfilterbank_setProperties#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  b_self = self;
  b_self->CutOff = 50.0;
  b_self->Gamma = 3.0;
  b_self->Beta = 20.0;
  b_self->Wavelet[0] = 'a';
  b_self->Wavelet[1] = 'm';
  b_self->Wavelet[2] = 'o';
  b_self->Wavelet[3] = 'r';
  b_self->TimeBandwidth = rtNaN;
  b_self->SignalLength = varargin_2;
  NyquistRange_idx_1 = b_self->SignalLength / 2.0;
  NyquistRange_idx_1 = std::floor(NyquistRange_idx_1);
  b_self->SignalPad = NyquistRange_idx_1;
  b_self->VoicesPerOctave = varargin_10;
  b_self->SamplingFrequency = varargin_8;
  b_self->WaveletParameters[0] = rtNaN;
  b_self->FrequencyLimits[0] = varargin_6[0];
  b_self->WaveletParameters[1] = rtNaN;
  b_self->FrequencyLimits[1] = varargin_6[1];
  profileLoopStart("cwtfilterbank_setProperties_loop_0", __LINE__, 9 + 1, "");
  for (k = 0; k < 10; k++) {
    b_self->Boundary[k] = b[k];
  }
  profileLoopEnd();
  NyquistRange_idx_1 = b_self->TimeBandwidth;
  guard1 = false;
  if (!std::isnan(NyquistRange_idx_1)) {
    NyquistRange_idx_1 = b_self->WaveletParameters[0];
    b_b[0] = std::isnan(NyquistRange_idx_1);
    NyquistRange_idx_1 = b_self->WaveletParameters[1];
    b_b[1] = std::isnan(NyquistRange_idx_1);
    freqsep = true;
    k = 0;
    exitg1 = false;
    nvtxRangePushA(
        "#loop#cwtfilterbank_setProperties_whileloop_5##" MW_AT_LINE);
    while ((!exitg1) && (k < 2)) {
      if (!b_b[k]) {
        freqsep = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    nvtxRangePop();
    if (freqsep) {
      b_self->Beta = b_self->TimeBandwidth / b_self->Gamma;
    } else {
      guard1 = true;
    }
  } else {
    guard1 = true;
  }
  if (guard1) {
    NyquistRange_idx_1 = b_self->WaveletParameters[0];
    b_b[0] = std::isnan(NyquistRange_idx_1);
    NyquistRange_idx_1 = b_self->WaveletParameters[1];
    b_b[1] = std::isnan(NyquistRange_idx_1);
    freqsep = true;
    k = 0;
    exitg1 = false;
    nvtxRangePushA(
        "#loop#cwtfilterbank_setProperties_whileloop_4##" MW_AT_LINE);
    while ((!exitg1) && (k < 2)) {
      if (!b_b[k]) {
        freqsep = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    nvtxRangePop();
    if (!freqsep) {
      NyquistRange_idx_1 = b_self->TimeBandwidth;
      if (std::isnan(NyquistRange_idx_1)) {
        b_self->Gamma = b_self->WaveletParameters[0];
        b_self->Beta = b_self->WaveletParameters[1] / b_self->Gamma;
      }
    }
  }
  profileLoopStart("cwtfilterbank_setProperties_loop_1", __LINE__, 9 + 1, "");
  for (k = 0; k < 10; k++) {
    a[k] = b_self->Boundary[k];
  }
  profileLoopEnd();
  freqsep = false;
  k = 0;
  nvtxRangePushA("#loop#cwtfilterbank_setProperties_whileloop_3##" MW_AT_LINE);
  do {
    exitg2 = 0;
    if (k + 1 < 11) {
      if (cv[static_cast<uint8_T>(a[k]) & 127] !=
          cv[static_cast<int32_T>(cv2[k])]) {
        exitg2 = 1;
      } else {
        k++;
      }
    } else {
      freqsep = true;
      exitg2 = 1;
    }
  } while (exitg2 == 0);
  nvtxRangePop();
  if (freqsep) {
    if (b_self->SignalLength <= 100000.0) {
      NyquistRange_idx_1 = b_self->SignalLength / 2.0;
      NyquistRange_idx_1 = std::floor(NyquistRange_idx_1);
      b_self->SignalPad = NyquistRange_idx_1;
    } else {
      nvtxMarkA("#b_log2#" MW_AT_LINE);
      NyquistRange_idx_1 = b_log2(b_self->SignalLength);
      NyquistRange_idx_1 = std::ceil(NyquistRange_idx_1);
      b_self->SignalPad = NyquistRange_idx_1;
    }
  } else {
    b_self->SignalPad = 0.0;
  }
  NyquistRange_idx_1 = b_self->FrequencyLimits[0];
  b_b[0] = std::isnan(NyquistRange_idx_1);
  NyquistRange_idx_1 = b_self->FrequencyLimits[1];
  b_b[1] = std::isnan(NyquistRange_idx_1);
  freqsep = true;
  k = 0;
  exitg1 = false;
  nvtxRangePushA("#loop#cwtfilterbank_setProperties_whileloop_2##" MW_AT_LINE);
  while ((!exitg1) && (k < 2)) {
    if (!b_b[k]) {
      freqsep = false;
      exitg1 = true;
    } else {
      k++;
    }
  }
  nvtxRangePop();
  if (!freqsep) {
    real_T FourierFactor;
    real_T be;
    real_T cutoff;
    real_T freqrange_idx_0;
    real_T freqrange_idx_1;
    real_T fs;
    real_T ga;
    real_T omegac;
    real_T varargin_1;
    real_T varargin_3;
    char_T b_wav[4];
    char_T wav[4];
    char_T c;
    freqrange_idx_0 = b_self->FrequencyLimits[0];
    freqrange_idx_1 = b_self->FrequencyLimits[1];
    NyquistRange_idx_1 = b_self->SamplingFrequency / 2.0;
    nvtxMarkA("#emxInit_char_T#" MW_AT_LINE);
    emxInit_char_T(&tmpStr, 2, &vc_emlrtRTEI, true);
    if ((freqrange_idx_1 <= 0.0) || (freqrange_idx_0 >= NyquistRange_idx_1)) {
      b_tmpStr = nullptr;
      y = nullptr;
      m = emlrtCreateCharArray(2, &iv[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &u[0]);
      emlrtAssign(&y, m);
      b_y = nullptr;
      m1 = emlrtCreateCharArray(2, &iv1[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 2, m1, &formatSpec[0]);
      emlrtAssign(&b_y, m1);
      c_y = nullptr;
      m2 = emlrtCreateDoubleScalar(NyquistRange_idx_1);
      emlrtAssign(&c_y, m2);
      nvtxMarkA("#feval#" MW_AT_LINE);
      emlrtAssign(&b_tmpStr, feval(y, b_y, c_y, &b_emlrtMCI));
      nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
      nvtxMarkA("#length#" MW_AT_LINE);
      emlrt_marshallIn(length(emlrtAlias(b_tmpStr), &c_emlrtMCI),
                       "<output of length>");
      nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
      emlrt_marshallIn(emlrtAlias(b_tmpStr), "tmpStr", tmpStr);
      emlrtDestroyArray(&b_tmpStr);
    }
    nvtxMarkA("#emxFree_char_T#" MW_AT_LINE);
    emxFree_char_T(&tmpStr);
    fs = b_self->SamplingFrequency;
    ga = b_self->Gamma;
    be = b_self->Beta;
    NyquistRange_idx_1 = b_self->SignalLength;
    varargin_3 = b_self->VoicesPerOctave;
    cutoff = b_self->CutOff;
    varargin_1 = b_self->SamplingFrequency;
    c = b_self->Wavelet[0];
    wav[0] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[1];
    wav[1] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[2];
    wav[2] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[3];
    wav[3] = cv[static_cast<uint8_T>(c) & 127];
    omegac = 3.1415926535897931;
    cutoff /= 100.0;
    b_wav[0] = wav[0];
    b_wav[1] = wav[1];
    b_wav[2] = wav[2];
    b_wav[3] = wav[3];
    nvtxMarkA("#wavCFandSD#" MW_AT_LINE);
    FourierFactor =
        wavelet::internal::cwt::wavCFandSD(b_wav, ga, be, &sigmat, &cf);
    sigmat = NyquistRange_idx_1 / (sigmat * 2.0);
    k = 0;
    nvtxRangePushA(
        "#loop#cwtfilterbank_setProperties_whileloop_1##" MW_AT_LINE);
    do {
      exitg2 = 0;
      if (k + 1 < 5) {
        if (cv3[k] != wav[k]) {
          exitg2 = 1;
        } else {
          k++;
        }
      } else {
        freqsep = true;
        exitg2 = 1;
      }
    } while (exitg2 == 0);
    nvtxRangePop();
    if (freqsep) {
      k = 1;
    } else {
      k = 0;
      nvtxRangePushA(
          "#loop#cwtfilterbank_setProperties_whileloop_0##" MW_AT_LINE);
      do {
        exitg2 = 0;
        if (k + 1 < 5) {
          if (cv4[k] != wav[k]) {
            exitg2 = 1;
          } else {
            k++;
          }
        } else {
          freqsep = true;
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      nvtxRangePop();
      if (freqsep) {
        k = 2;
      } else {
        k = -1;
      }
    }
    switch (k) {
    case 0:
      nvtxMarkA("#getFreqFromCutoffMorse#" MW_AT_LINE);
      omegac =
          wavelet::internal::cwt::getFreqFromCutoffMorse(cutoff, cf, ga, be);
      break;
    case 1:
      nvtxMarkA("#getFreqFromCutoffBump#" MW_AT_LINE);
      omegac = wavelet::internal::cwt::getFreqFromCutoffBump(cutoff, cf);
      break;
    case 2:
      nvtxMarkA("#getFreqFromCutoffAmor#" MW_AT_LINE);
      omegac = wavelet::internal::cwt::getFreqFromCutoffAmor(cutoff, cf);
      break;
    }
    NyquistRange_idx_1 = omegac / 3.1415926535897931;
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    if (sigmat < NyquistRange_idx_1 * rt_powd_snf(2.0, 1.0 / varargin_3)) {
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      sigmat = NyquistRange_idx_1 * rt_powd_snf(2.0, 1.0 / varargin_3);
    }
    NyquistRange_idx_1 = 1.0 / (sigmat * FourierFactor) * varargin_1;
    if (freqrange_idx_0 < NyquistRange_idx_1) {
      b_self->FrequencyLimits[0] = NyquistRange_idx_1;
      freqrange_idx_0 = b_self->FrequencyLimits[0];
    }
    if (freqrange_idx_1 > fs / 2.0) {
      b_self->FrequencyLimits[1] = fs / 2.0;
      freqrange_idx_1 = b_self->FrequencyLimits[1];
    }
    nvtxMarkA("#b_log2#" MW_AT_LINE);
    nvtxMarkA("#b_log2#" MW_AT_LINE);
    freqsep = (b_log2(freqrange_idx_1) - b_log2(freqrange_idx_0) >=
               1.0 / b_self->VoicesPerOctave);
    if (!freqsep) {
      varargin_1 = 1.0 / b_self->VoicesPerOctave;
      c_tmpStr = nullptr;
      d_y = nullptr;
      m3 = emlrtCreateCharArray(2, &iv2[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m3, &u[0]);
      emlrtAssign(&d_y, m3);
      e_y = nullptr;
      m4 = emlrtCreateCharArray(2, &iv3[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m4, &b_formatSpec[0]);
      emlrtAssign(&e_y, m4);
      f_y = nullptr;
      m5 = emlrtCreateDoubleScalar(varargin_1);
      emlrtAssign(&f_y, m5);
      nvtxMarkA("#feval#" MW_AT_LINE);
      emlrtAssign(&c_tmpStr, feval(d_y, e_y, f_y, &b_emlrtMCI));
      nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
      nvtxMarkA("#length#" MW_AT_LINE);
      emlrt_marshallIn(length(emlrtAlias(c_tmpStr), &c_emlrtMCI),
                       "<output of length>");
      nvtxMarkA("#emxInit_char_T#" MW_AT_LINE);
      emxInit_char_T(&d_tmpStr, 2, &vc_emlrtRTEI, true);
      nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
      emlrt_marshallIn(emlrtAlias(c_tmpStr), "tmpStr", d_tmpStr);
      nvtxMarkA("#emxFree_char_T#" MW_AT_LINE);
      emxFree_char_T(&d_tmpStr);
      emlrtDestroyArray(&c_tmpStr);
    }
  }
  b_self->CutOff = 10.0;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
  return b_self;
}

} // namespace coder
static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y)
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(tmpStr), &thisId, y);
  emlrtDestroyArray(&tmpStr);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y)
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location)
{
  const mxArray *pArrays[3];
  const mxArray *m;
  const mxArray *m4;
  nvtxRangePushA("#fcn#feval#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  pArrays[2] = m3;
  m4 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 3, &pArrays[0], "feval",
                             true, location);
  nvtxRangePop();
  return m4;
}

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location)
{
  const mxArray *m;
  const mxArray *m2;
  const mxArray *pArray;
  nvtxRangePushA("#fcn#length#" MW_AT_LOCATION);
  pArray = m1;
  m2 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 1, &pArray, "length",
                             true, location);
  nvtxRangePop();
  return m2;
}

//
//
namespace coder {
cwtfilterbank *cwtfilterbank_cwtfilterbank(cwtfilterbank *self,
                                           real_T varargin_2,
                                           const real_T varargin_6[2],
                                           real_T varargin_8,
                                           real_T varargin_10)
{
  static const char_T cv1[4]{'b', 'u', 'm', 'p'};
  static const char_T cv2[4]{'a', 'm', 'o', 'r'};
  cwtfilterbank *b_self;
  emxArray_real_T *b_y;
  emxArray_real_T *c_b;
  emxArray_real_T *c_y;
  emxArray_real_T *d_b;
  emxArray_real_T *d_y;
  emxArray_real_T *daughter;
  emxArray_real_T *expnt;
  emxArray_real_T *f;
  emxArray_real_T *omega;
  emxArray_real_T *omega_tmp1;
  emxArray_real_T *omega_tmp2;
  emxArray_real_T *scales;
  emxArray_real_T *somega;
  emxArray_real_T *w;
  real_T N;
  real_T b;
  real_T cf;
  real_T ga;
  real_T nbSamp;
  int32_T b_i;
  int32_T exitg2;
  int32_T fc;
  int32_T i;
  int32_T loop_ub;
  int32_T nx;
  char_T wname[4];
  boolean_T b_b[2];
  boolean_T exitg1;
  boolean_T y;
  nvtxRangePushA("#fcn#cwtfilterbank_cwtfilterbank#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  b_self = self;
  nvtxMarkA("#cwtfilterbank_setProperties#" MW_AT_LINE);
  b_self = cwtfilterbank_setProperties(b_self, varargin_2, varargin_6,
                                       varargin_8, varargin_10);
  N = b_self->SignalLength + 2.0 * b_self->SignalPad;
  b = std::trunc(N / 2.0);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&omega_tmp1, 2, &wb_emlrtRTEI, true);
  if (std::isnan(b)) {
    i = omega_tmp1->size[0] * omega_tmp1->size[1];
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(omega_tmp1, i, &wb_emlrtRTEI);
    omega_tmp1->data[0] = rtNaN;
  } else if (b < 1.0) {
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = 0;
  } else {
    i = omega_tmp1->size[0] * omega_tmp1->size[1];
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = static_cast<int32_T>(b - 1.0) + 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(omega_tmp1, i, &wb_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_0", __LINE__,
                     static_cast<int32_T>(b - 1.0) + 1, "");
    for (i = 0; i <= static_cast<int32_T>(b - 1.0); i++) {
      omega_tmp1->data[i] = static_cast<real_T>(i) + 1.0;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&omega_tmp2, 2, &xb_emlrtRTEI, true);
  i = omega_tmp2->size[0] * omega_tmp2->size[1];
  omega_tmp2->size[0] = 1;
  omega_tmp2->size[1] = omega_tmp1->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(omega_tmp2, i, &xb_emlrtRTEI);
  nbSamp = 6.2831853071795862 / N;
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_1", __LINE__,
                   (omega_tmp1->size[1] - 1) + 1, "");
  for (i = 0; i < omega_tmp1->size[1]; i++) {
    omega_tmp2->data[i] = omega_tmp1->data[i] * nbSamp;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&omega_tmp1);
  N = std::trunc((N - 1.0) / 2.0);
  if (N < 1.0) {
    b_i = 0;
    nx = 1;
    fc = -1;
  } else {
    b_i = static_cast<int32_T>(N) - 1;
    nx = -1;
    fc = 0;
  }
  i = b_self->Omega->size[0] * b_self->Omega->size[1];
  b_self->Omega->size[0] = 1;
  nvtxMarkA("#div_s32#" MW_AT_LINE);
  b_self->Omega->size[1] = (omega_tmp2->size[1] + div_s32(fc - b_i, nx)) + 2;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_self->Omega, i, &yb_emlrtRTEI);
  b_self->Omega->data[0] = 0.0;
  loop_ub = omega_tmp2->size[1];
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_2", __LINE__,
                   (loop_ub - 1) + 1, "");
  for (i = 0; i < loop_ub; i++) {
    b_self->Omega->data[i + 1] = omega_tmp2->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#div_s32#" MW_AT_LINE);
  loop_ub = div_s32(fc - b_i, nx);
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_3", __LINE__, loop_ub + 1,
                   "");
  for (i = 0; i <= loop_ub; i++) {
    b_self->Omega->data[(i + omega_tmp2->size[1]) + 1] =
        -omega_tmp2->data[b_i + nx * i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&omega_tmp2);
  nbSamp = b_self->FrequencyLimits[0];
  b_b[0] = std::isnan(nbSamp);
  nbSamp = b_self->FrequencyLimits[1];
  b_b[1] = std::isnan(nbSamp);
  y = true;
  loop_ub = 0;
  exitg1 = false;
  nvtxRangePushA("#loop#cwtfilterbank_cwtfilterbank_whileloop_3##" MW_AT_LINE);
  while ((!exitg1) && (loop_ub < 2)) {
    if (!b_b[loop_ub]) {
      y = false;
      exitg1 = true;
    } else {
      loop_ub++;
    }
  }
  nvtxRangePop();
  if (!y) {
    real_T frange_idx_0;
    real_T nv;
    char_T b_wname[4];
    nbSamp = b_self->FrequencyLimits[0];
    nbSamp /= b_self->SamplingFrequency;
    nbSamp = nbSamp * 2.0 * 3.1415926535897931;
    frange_idx_0 = nbSamp;
    nbSamp = b_self->FrequencyLimits[1];
    nbSamp /= b_self->SamplingFrequency;
    nbSamp = nbSamp * 2.0 * 3.1415926535897931;
    nv = b_self->VoicesPerOctave;
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    ga = rt_powd_snf(2.0, 1.0 / nv);
    b_wname[0] = b_self->Wavelet[0];
    b_wname[1] = b_self->Wavelet[1];
    b_wname[2] = b_self->Wavelet[2];
    b_wname[3] = b_self->Wavelet[3];
    nvtxMarkA("#wavCFandSD#" MW_AT_LINE);
    wavelet::internal::cwt::wavCFandSD(b_wname, b_self->Gamma, b_self->Beta, &b,
                                       &cf);
    N = cf / nbSamp;
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_y, 2, &oc_emlrtRTEI, true);
    nvtxMarkA("#b_log2#" MW_AT_LINE);
    b = nv * b_log2(cf / frange_idx_0 / N);
    if (std::isnan(b)) {
      i = b_y->size[0] * b_y->size[1];
      b_y->size[0] = 1;
      b_y->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_y, i, &n_emlrtRTEI);
      b_y->data[0] = rtNaN;
    } else if (b < 0.0) {
      b_y->size[0] = 1;
      b_y->size[1] = 0;
    } else {
      i = b_y->size[0] * b_y->size[1];
      b_y->size[0] = 1;
      b_y->size[1] = static_cast<int32_T>(b) + 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_y, i, &n_emlrtRTEI);
      profileLoopStart("cwtfilterbank_cwtfilterbank_loop_4", __LINE__,
                       static_cast<int32_T>(b) + 1, "");
      for (i = 0; i <= static_cast<int32_T>(b); i++) {
        b_y->data[i] = i;
      }
      profileLoopEnd();
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_b, 2, &pc_emlrtRTEI, true);
    i = c_b->size[0] * c_b->size[1];
    c_b->size[0] = 1;
    c_b->size[1] = b_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(c_b, i, &kb_emlrtRTEI);
    nx = b_y->size[1];
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_5", __LINE__,
                     (nx - 1) + 1, "");
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      c_b->data[loop_ub] = rt_powd_snf(ga, b_y->data[loop_ub]);
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_y);
    i = b_self->Scales->size[0] * b_self->Scales->size[1];
    b_self->Scales->size[0] = 1;
    b_self->Scales->size[1] = c_b->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b_self->Scales, i, &ac_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_6", __LINE__,
                     (c_b->size[1] - 1) + 1, "");
    for (i = 0; i < c_b->size[1]; i++) {
      b_self->Scales->data[i] = N * c_b->data[i];
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_b);
  } else {
    real_T be;
    real_T frange_idx_0;
    real_T nv;
    real_T omegac;
    char_T b_wname[4];
    char_T c;
    nbSamp = b_self->SignalLength;
    ga = b_self->Gamma;
    be = b_self->Beta;
    nv = b_self->VoicesPerOctave;
    N = b_self->CutOff;
    c = b_self->Wavelet[0];
    wname[0] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[1];
    wname[1] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[2];
    wname[2] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[3];
    wname[3] = cv[static_cast<uint8_T>(c) & 127];
    omegac = 3.1415926535897931;
    frange_idx_0 = N / 100.0;
    b_wname[0] = wname[0];
    b_wname[1] = wname[1];
    b_wname[2] = wname[2];
    b_wname[3] = wname[3];
    nvtxMarkA("#wavCFandSD#" MW_AT_LINE);
    wavelet::internal::cwt::wavCFandSD(b_wname, ga, be, &b, &cf);
    b = nbSamp / (b * 2.0);
    y = false;
    nx = 0;
    nvtxRangePushA(
        "#loop#cwtfilterbank_cwtfilterbank_whileloop_2##" MW_AT_LINE);
    do {
      exitg2 = 0;
      if (nx + 1 < 5) {
        if (cv1[nx] != wname[nx]) {
          exitg2 = 1;
        } else {
          nx++;
        }
      } else {
        y = true;
        exitg2 = 1;
      }
    } while (exitg2 == 0);
    nvtxRangePop();
    if (y) {
      nx = 1;
    } else {
      nx = 0;
      nvtxRangePushA(
          "#loop#cwtfilterbank_cwtfilterbank_whileloop_1##" MW_AT_LINE);
      do {
        exitg2 = 0;
        if (nx + 1 < 5) {
          if (cv2[nx] != wname[nx]) {
            exitg2 = 1;
          } else {
            nx++;
          }
        } else {
          y = true;
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      nvtxRangePop();
      if (y) {
        nx = 2;
      } else {
        nx = -1;
      }
    }
    switch (nx) {
    case 0:
      nvtxMarkA("#getFreqFromCutoffMorse#" MW_AT_LINE);
      omegac = wavelet::internal::cwt::getFreqFromCutoffMorse(frange_idx_0, cf,
                                                              ga, be);
      break;
    case 1:
      nvtxMarkA("#getFreqFromCutoffBump#" MW_AT_LINE);
      omegac = wavelet::internal::cwt::getFreqFromCutoffBump(frange_idx_0, cf);
      break;
    case 2:
      nvtxMarkA("#getFreqFromCutoffAmor#" MW_AT_LINE);
      omegac = wavelet::internal::cwt::getFreqFromCutoffAmor(frange_idx_0, cf);
      break;
    }
    N = omegac / 3.1415926535897931;
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    if (b < N * rt_powd_snf(2.0, 1.0 / nv)) {
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      b = N * rt_powd_snf(2.0, 1.0 / nv);
    }
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    ga = rt_powd_snf(2.0, 1.0 / nv);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_y, 2, &rc_emlrtRTEI, true);
    nvtxMarkA("#b_log2#" MW_AT_LINE);
    b = std::fmax(b_log2(b / N), 1.0 / nv) * nv;
    if (std::isnan(b)) {
      i = c_y->size[0] * c_y->size[1];
      c_y->size[0] = 1;
      c_y->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(c_y, i, &n_emlrtRTEI);
      c_y->data[0] = rtNaN;
    } else if (b < 0.0) {
      c_y->size[0] = 1;
      c_y->size[1] = 0;
    } else {
      i = c_y->size[0] * c_y->size[1];
      c_y->size[0] = 1;
      c_y->size[1] = static_cast<int32_T>(b) + 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(c_y, i, &n_emlrtRTEI);
      profileLoopStart("cwtfilterbank_cwtfilterbank_loop_7", __LINE__,
                       static_cast<int32_T>(b) + 1, "");
      for (i = 0; i <= static_cast<int32_T>(b); i++) {
        c_y->data[i] = i;
      }
      profileLoopEnd();
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&d_b, 2, &sc_emlrtRTEI, true);
    i = d_b->size[0] * d_b->size[1];
    d_b->size[0] = 1;
    d_b->size[1] = c_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(d_b, i, &kb_emlrtRTEI);
    nx = c_y->size[1];
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_9", __LINE__,
                     (nx - 1) + 1, "");
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      d_b->data[loop_ub] = rt_powd_snf(ga, c_y->data[loop_ub]);
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_y);
    i = b_self->Scales->size[0] * b_self->Scales->size[1];
    b_self->Scales->size[0] = 1;
    b_self->Scales->size[1] = d_b->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b_self->Scales, i, &dc_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_11", __LINE__,
                     (d_b->size[1] - 1) + 1, "");
    for (i = 0; i < d_b->size[1]; i++) {
      b_self->Scales->data[i] = N * d_b->data[i];
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&d_b);
  }
  wname[0] = b_self->Wavelet[0];
  wname[1] = b_self->Wavelet[1];
  wname[2] = b_self->Wavelet[2];
  wname[3] = b_self->Wavelet[3];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&omega, 2, &bc_emlrtRTEI, true);
  i = omega->size[0] * omega->size[1];
  omega->size[0] = 1;
  omega->size[1] = b_self->Omega->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(omega, i, &bc_emlrtRTEI);
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_8", __LINE__,
                   (b_self->Omega->size[1] - 1) + 1, "");
  for (i = 0; i < b_self->Omega->size[1]; i++) {
    omega->data[i] = b_self->Omega->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&scales, 2, &cc_emlrtRTEI, true);
  i = scales->size[0] * scales->size[1];
  scales->size[0] = 1;
  scales->size[1] = b_self->Scales->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(scales, i, &cc_emlrtRTEI);
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_10", __LINE__,
                   (b_self->Scales->size[1] - 1) + 1, "");
  for (i = 0; i < b_self->Scales->size[1]; i++) {
    scales->data[i] = b_self->Scales->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&somega, 2, &qc_emlrtRTEI, true);
  if (scales->size[1] == 1) {
    i = somega->size[0] * somega->size[1];
    somega->size[0] = 1;
    if (omega->size[1] == 1) {
      somega->size[1] = 1;
    } else {
      somega->size[1] = omega->size[1];
    }
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(somega, i, &fc_emlrtRTEI);
    if (somega->size[1] != 0) {
      nx = (omega->size[1] != 1);
      fc = somega->size[1] - 1;
      profileLoopStart("cwtfilterbank_cwtfilterbank_loop_14", __LINE__, fc + 1,
                       "");
      for (loop_ub = 0; loop_ub <= fc; loop_ub++) {
        somega->data[loop_ub] = scales->data[0] * omega->data[nx * loop_ub];
      }
      profileLoopEnd();
    }
  } else {
    i = somega->size[0] * somega->size[1];
    somega->size[0] = scales->size[1];
    somega->size[1] = omega->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(somega, i, &ec_emlrtRTEI);
    loop_ub = omega->size[1];
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_12", __LINE__,
                     (loop_ub - 1) + 1, "");
    for (i = 0; i < loop_ub; i++) {
      fc = scales->size[1];
      profileLoopStart("cwtfilterbank_cwtfilterbank_loop_13", __LINE__,
                       (fc - 1) + 1, "");
      for (nx = 0; nx < fc; nx++) {
        somega->data[nx + somega->size[0] * i] =
            scales->data[nx] * omega->data[i];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&omega);
  y = false;
  nx = 0;
  nvtxRangePushA("#loop#cwtfilterbank_cwtfilterbank_whileloop_0##" MW_AT_LINE);
  do {
    exitg2 = 0;
    if (nx + 1 < 5) {
      if (cv2[nx] != wname[nx]) {
        exitg2 = 1;
      } else {
        nx++;
      }
    } else {
      y = true;
      exitg2 = 1;
    }
  } while (exitg2 == 0);
  nvtxRangePop();
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&expnt, 2, &hc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&daughter, 2, &mc_emlrtRTEI, true);
  if (y) {
    fc = 0;
  } else {
    fc = -1;
  }
  if (fc == 0) {
    fc = 6;
    i = expnt->size[0] * expnt->size[1];
    expnt->size[0] = somega->size[0];
    expnt->size[1] = somega->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(expnt, i, &hc_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_15", __LINE__,
                     (somega->size[0] * somega->size[1] - 1) + 1, "");
    for (i = 0; i < somega->size[0] * somega->size[1]; i++) {
      expnt->data[i] = -((somega->data[i] - 6.0) * (somega->data[i] - 6.0)) /
                       2.0 * static_cast<real_T>(somega->data[i] > 0.0);
    }
    profileLoopEnd();
    nx = expnt->size[0] * expnt->size[1];
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_17", __LINE__,
                     (nx - 1) + 1, "");
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      expnt->data[loop_ub] = std::exp(expnt->data[loop_ub]);
    }
    profileLoopEnd();
    i = daughter->size[0] * daughter->size[1];
    daughter->size[0] = expnt->size[0];
    daughter->size[1] = expnt->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(daughter, i, &jc_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_19", __LINE__,
                     (expnt->size[0] * expnt->size[1] - 1) + 1, "");
    for (i = 0; i < expnt->size[0] * expnt->size[1]; i++) {
      daughter->data[i] =
          2.0 * expnt->data[i] * static_cast<real_T>(somega->data[i] > 0.0);
    }
    profileLoopEnd();
  } else {
    fc = 5;
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&w, 2, &gc_emlrtRTEI, true);
    i = w->size[0] * w->size[1];
    w->size[0] = somega->size[0];
    w->size[1] = somega->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(w, i, &gc_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_16", __LINE__,
                     (somega->size[0] * somega->size[1] - 1) + 1, "");
    for (i = 0; i < somega->size[0] * somega->size[1]; i++) {
      w->data[i] = (somega->data[i] - 5.0) / 0.6;
    }
    profileLoopEnd();
    i = expnt->size[0] * expnt->size[1];
    expnt->size[0] = w->size[0];
    expnt->size[1] = w->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(expnt, i, &ic_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_18", __LINE__,
                     (w->size[0] * w->size[1] - 1) + 1, "");
    for (i = 0; i < w->size[0] * w->size[1]; i++) {
      expnt->data[i] = -1.0 / (1.0 - w->data[i] * w->data[i]);
    }
    profileLoopEnd();
    nx = expnt->size[0] * expnt->size[1];
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_20", __LINE__,
                     (nx - 1) + 1, "");
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      expnt->data[loop_ub] = std::exp(expnt->data[loop_ub]);
    }
    profileLoopEnd();
    nx = w->size[0] * w->size[1];
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&d_y, 2, &tc_emlrtRTEI, true);
    i = d_y->size[0] * d_y->size[1];
    d_y->size[0] = w->size[0];
    d_y->size[1] = w->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(d_y, i, &h_emlrtRTEI);
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_21", __LINE__,
                     (nx - 1) + 1, "");
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      d_y->data[loop_ub] = std::abs(w->data[loop_ub]);
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&w);
    if ((expnt->size[0] == d_y->size[0]) && (expnt->size[1] == d_y->size[1])) {
      i = daughter->size[0] * daughter->size[1];
      daughter->size[0] = expnt->size[0];
      daughter->size[1] = expnt->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(daughter, i, &mc_emlrtRTEI);
      profileLoopStart("cwtfilterbank_cwtfilterbank_loop_24", __LINE__,
                       (expnt->size[0] * expnt->size[1] - 1) + 1, "");
      for (i = 0; i < expnt->size[0] * expnt->size[1]; i++) {
        daughter->data[i] =
            5.43656365691809 * expnt->data[i] *
            static_cast<real_T>(d_y->data[i] < 0.99999999999999978);
      }
      profileLoopEnd();
    } else {
      nvtxMarkA("#binary_expand_op_2#" MW_AT_LINE);
      binary_expand_op_2(daughter, expnt, d_y);
    }
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&d_y);
    nx = daughter->size[0] * daughter->size[1] - 1;
    profileLoopStart("cwtfilterbank_cwtfilterbank_loop_23", __LINE__, nx + 1,
                     "");
    for (b_i = 0; b_i <= nx; b_i++) {
      if (std::isnan(daughter->data[b_i])) {
        daughter->data[b_i] = 0.0;
      }
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&expnt);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&somega);
  ga = static_cast<real_T>(fc) / 6.2831853071795862;
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&f, 2, &uc_emlrtRTEI, true);
  i = f->size[0] * f->size[1];
  f->size[0] = 1;
  f->size[1] = scales->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(f, i, &kc_emlrtRTEI);
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_22", __LINE__,
                   (scales->size[1] - 1) + 1, "");
  for (i = 0; i < scales->size[1]; i++) {
    f->data[i] = ga / scales->data[i] * b_self->SamplingFrequency;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&scales);
  i = b_self->PsiDFT->size[0] * b_self->PsiDFT->size[1];
  b_self->PsiDFT->size[0] = daughter->size[0];
  b_self->PsiDFT->size[1] = daughter->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_self->PsiDFT, i, &lc_emlrtRTEI);
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_25", __LINE__,
                   (daughter->size[0] * daughter->size[1] - 1) + 1, "");
  for (i = 0; i < daughter->size[0] * daughter->size[1]; i++) {
    b_self->PsiDFT->data[i] = daughter->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&daughter);
  i = b_self->WaveletCenterFrequencies->size[0];
  b_self->WaveletCenterFrequencies->size[0] = f->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_self->WaveletCenterFrequencies, i, &nc_emlrtRTEI);
  profileLoopStart("cwtfilterbank_cwtfilterbank_loop_26", __LINE__,
                   (f->size[1] - 1) + 1, "");
  for (i = 0; i < f->size[1]; i++) {
    b_self->WaveletCenterFrequencies->data[i] = f->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&f);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
  return b_self;
}

} // namespace coder

// End of code generation (cwtfilterbank.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ifft.cu
//
// Code generation for function 'ifft'
//

// Include files
#include "ifft.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hipfft/hipfft.h"
#include "nvtx3/nvToolsExt.h"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo vd_emlrtRTEI{
    83,                                                              // lineNo
    1,                                                               // colNo
    "ifft",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/ifft.m" // pName
};

// Function Definitions
//
//
namespace coder {
void ifft(const emxArray_creal_T *x, emxArray_creal_T *y)
{
  emxArray_creal_T gpu_x;
  emxArray_creal_T *cpu_x;
  nvtxRangePushA("#fcn#ifft#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_x);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((x->size[0] == 0) || (x->size[1] == 0)) {
    int32_T i;
    uint32_T ySize_idx_0;
    uint32_T ySize_idx_1;
    ySize_idx_0 = static_cast<uint32_T>(x->size[0]);
    ySize_idx_1 = static_cast<uint32_T>(x->size[1]);
    i = y->size[0] * y->size[1];
    y->size[0] = x->size[0];
    y->size[1] = x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(y, i, &vd_emlrtRTEI);
    if (static_cast<int32_T>(ySize_idx_0) * static_cast<int32_T>(ySize_idx_1) -
            1 >=
        0) {
      std::memset(&y->data[0], 0,
                  static_cast<uint32_T>(static_cast<int32_T>(ySize_idx_0) *
                                        static_cast<int32_T>(ySize_idx_1)) *
                      sizeof(creal_T));
    }
  } else {
    hipfftHandle fftPlanHandle;
    int32_T i;
    int32_T inembed;
    boolean_T x_outdatedOnGpu;
    nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
    emxInit_creal_T(&cpu_x, 2, &ud_emlrtRTEI, true);
    x_outdatedOnGpu = false;
    i = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = x->size[0];
    cpu_x->size[1] = x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_x, i, &ud_emlrtRTEI);
    profileLoopStart("ifft_loop_0", __LINE__, (x->size[0] * x->size[1] - 1) + 1,
                     "");
    for (i = 0; i < x->size[0] * x->size[1]; i++) {
      cpu_x->data[i] = x->data[i];
      x_outdatedOnGpu = true;
    }
    profileLoopEnd();
    inembed = x->size[1];
    fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, x->size[0], 1,
                                     HIPFFT_Z2Z, x->size[0]);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    if (x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_x, cpu_x);
    }
    hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&gpu_x.data[0],
                 (hipfftDoubleComplex *)&gpu_x.data[0], HIPFFT_BACKWARD);
    x_outdatedOnGpu = true;
    i = y->size[0] * y->size[1];
    y->size[0] = cpu_x->size[0];
    y->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(y, i, &vd_emlrtRTEI);
    inembed = x->size[1];
    profileLoopStart("ifft_loop_1", __LINE__,
                     (cpu_x->size[0] * cpu_x->size[1] - 1) + 1, "");
    for (i = 0; i < cpu_x->size[0] * cpu_x->size[1]; i++) {
      real_T ai;
      real_T ar;
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_creal_T(cpu_x, &gpu_x);
      }
      ar = cpu_x->data[i].re;
      x_outdatedOnGpu = false;
      ai = cpu_x->data[i].im;
      if (ai == 0.0) {
        y->data[i].re = ar / static_cast<real_T>(inembed);
        y->data[i].im = 0.0;
      } else if (ar == 0.0) {
        y->data[i].re = 0.0;
        y->data[i].im = ai / static_cast<real_T>(inembed);
      } else {
        y->data[i].re = ar / static_cast<real_T>(inembed);
        y->data[i].im = ai / static_cast<real_T>(inembed);
      }
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
    emxFree_creal_T(&cpu_x);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_x);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (ifft.cu)

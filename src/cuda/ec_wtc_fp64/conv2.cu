#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// conv2.cu
//
// Code generation for function 'conv2'
//

// Include files
#include "conv2.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "mtimes.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo xf_emlrtRTEI{
    56,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo yf_emlrtRTEI{
    47,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo bg_emlrtRTEI{
    49,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo jg_emlrtRTEI{
    58,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

// Function Declarations
namespace coder {
static void
b_conv2NonSeparable(emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
                    emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
                    emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
                    emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
                    emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu);

static void
c_conv2NonSeparable(emxArray_creal_T *cpu_a, emxArray_creal_T *gpu_a,
                    boolean_T *a_outdatedOnGpu, const emxArray_real_T *b,
                    emxArray_creal_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_creal_T *gpu_c, boolean_T *c_outdatedOnGpu);

static void
conv2NonSeparable(emxArray_real_T *cpu_a, emxArray_real_T *gpu_a,
                  boolean_T *a_outdatedOnGpu, emxArray_real_T *cpu_b,
                  boolean_T *b_outdatedOnCpu, emxArray_real_T *gpu_b,
                  boolean_T *b_outdatedOnGpu, emxArray_real_T *cpu_c,
                  boolean_T *c_outdatedOnCpu, emxArray_real_T *gpu_c,
                  boolean_T *c_outdatedOnGpu);

} // namespace coder
static __global__ void conv2NonSeparable_kernel119(const int32_T b,
                                                   emxArray_creal_T c);

static __global__ void conv2NonSeparable_kernel120(const int32_T b,
                                                   emxArray_creal_T c);

static __global__ void conv2NonSeparable_kernel121(const real_T cv_re,
                                                   creal_T b_data[1]);

static __global__ void conv2NonSeparable_kernel122(const emxArray_creal_T a,
                                                   const creal_T b_data[1],
                                                   emxArray_creal_T b_a);

static __global__ void conv2NonSeparable_kernel123(const emxArray_creal_T a,
                                                   const int32_T iv,
                                                   emxArray_creal_T c);

static __global__ void conv2NonSeparable_kernel124(const int32_T OH,
                                                   emxArray_creal_T expanded);

static __global__ void conv2NonSeparable_kernel125(const int32_T offsetH,
                                                   const emxArray_int32_T y,
                                                   const int32_T b_y,
                                                   emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel126(const emxArray_int32_T y,
                                                   const int32_T b_y,
                                                   emxArray_int32_T iv1);

static __global__ void conv2NonSeparable_kernel127(
    const emxArray_creal_T a, const int32_T iv_dim0, const emxArray_int32_T iv1,
    const emxArray_int32_T iv, const int32_T b_iv, const int32_T c_iv,
    emxArray_creal_T expanded, int32_T expanded_dim0);

static __global__ void
conv2NonSeparable_kernel128(const int32_T offsetH,
                            const emxArray_creal_T expanded, const int32_T k,
                            const emxArray_int32_T rows, const int32_T b_rows,
                            emxArray_creal_T newIm, int32_T expanded_dim0);

static __global__ void conv2NonSeparable_kernel129(const real_T cv_re,
                                                   const int32_T offsetH,
                                                   const int32_T k,
                                                   emxArray_creal_T c,
                                                   int32_T c_dim0);

static __global__ void conv2NonSeparable_kernel130(const real_T cv_im,
                                                   const int32_T offsetH,
                                                   const int32_T k,
                                                   emxArray_creal_T c,
                                                   int32_T c_dim0);

static __global__ void conv2NonSeparable_kernel58(const int32_T b,
                                                  emxArray_real_T c);

static __global__ void conv2NonSeparable_kernel59(const int32_T b,
                                                  emxArray_real_T c);

static __global__ void conv2NonSeparable_kernel60(const emxArray_real_T c,
                                                  const int32_T b,
                                                  emxArray_real_T b_c);

static __global__ void conv2NonSeparable_kernel61(const int32_T OH,
                                                  emxArray_real_T expanded);

static __global__ void conv2NonSeparable_kernel62(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel63(const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv1);

static __global__ void conv2NonSeparable_kernel64(
    const emxArray_real_T a, const int32_T iv_dim0, const emxArray_int32_T iv1,
    const emxArray_int32_T iv, const int32_T b_iv, const int32_T c_iv,
    emxArray_real_T expanded, int32_T expanded_dim0);

static __global__ void conv2NonSeparable_kernel65(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_real_T b, const int32_T c, const int32_T d,
    emxArray_real_T b_c, int32_T b_dim0, int32_T expanded_dim0, int32_T c_dim0);

static __global__ void conv2NonSeparable_kernel66(const int32_T b,
                                                  emxArray_real_T c);

static __global__ void conv2NonSeparable_kernel67(const int32_T OH,
                                                  emxArray_real_T expanded);

static __global__ void conv2NonSeparable_kernel68(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel69(const emxArray_real_T a,
                                                  const emxArray_int32_T iv,
                                                  const int32_T b,
                                                  emxArray_real_T expanded);

static __global__ void
conv2NonSeparable_kernel70(const emxArray_real_T expanded,
                           const emxArray_int32_T rows, const emxArray_real_T b,
                           const int32_T c, emxArray_real_T b_c,
                           int32_T b_dim0);

static __global__ void conv2_kernel102(const emxArray_creal_T a,
                                       const int32_T b_a, emxArray_creal_T c_a,
                                       int32_T a_dim0);

static __global__ void conv2_kernel103(const int32_T a, emxArray_creal_T c);

static __global__ void conv2_kernel104(const emxArray_creal_T a, const real_T b,
                                       real_T *d1, real_T *d);

static __global__ void conv2_kernel105(const real_T *d, const real_T *d1,
                                       emxArray_creal_T c);

static __global__ void conv2_kernel106(const emxArray_creal_T c,
                                       const int32_T b_c, emxArray_creal_T c_c,
                                       int32_T c_dim0);

static __global__ void conv2_kernel107(const int32_T a,
                                       emxArray_creal_T expanded);

static __global__ void conv2_kernel108(const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel109(const emxArray_creal_T a,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_creal_T expanded);

static __global__ void conv2_kernel110(const emxArray_creal_T expanded,
                                       const real_T b, const int32_T c,
                                       emxArray_creal_T b_c);

static __global__ void conv2_kernel111(const int32_T b, emxArray_creal_T c);

static __global__ void conv2_kernel112(const int32_T b, emxArray_creal_T c);

static __global__ void conv2_kernel113(const real_T d, creal_T b_data[1]);

static __global__ void conv2_kernel114(const emxArray_creal_T a,
                                       const creal_T b_data[1],
                                       emxArray_creal_T c);

static __global__ void conv2_kernel115(const int32_T OH,
                                       emxArray_real_T expanded);

static __global__ void conv2_kernel116(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv1);

static __global__ void
conv2_kernel117(const emxArray_real_T b, const int32_T mc,
                const emxArray_int32_T iv1, const int32_T c,
                emxArray_real_T expanded, int32_T expanded_dim0);

static __global__ void
conv2_kernel118(const emxArray_real_T expanded, const emxArray_int32_T cols,
                const emxArray_int32_T rows, const emxArray_creal_T a,
                const int32_T b, const int32_T c, emxArray_creal_T b_c,
                int32_T a_dim0, int32_T a_dim1, int32_T expanded_dim0,
                int32_T c_dim0);

static __global__ void conv2_kernel131(const int32_T b, emxArray_creal_T c);

static __global__ void conv2_kernel132(const int32_T OH,
                                       emxArray_real_T expanded);

static __global__ void conv2_kernel133(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv1);

static __global__ void conv2_kernel134(const emxArray_real_T b,
                                       const emxArray_int32_T iv1,
                                       const int32_T c,
                                       emxArray_real_T expanded);

static __global__ void conv2_kernel135(const emxArray_real_T expanded,
                                       const emxArray_int32_T rows,
                                       const emxArray_creal_T a,
                                       const int32_T b, emxArray_creal_T c,
                                       int32_T a_dim0);

static __global__ void conv2_kernel136(const int32_T b, emxArray_creal_T c);

static __global__ void conv2_kernel137(const int32_T OH,
                                       emxArray_creal_T expanded);

static __global__ void conv2_kernel138(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel139(const emxArray_creal_T a,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_creal_T expanded);

static __global__ void conv2_kernel140(const emxArray_creal_T expanded,
                                       const int32_T offsetH,
                                       const emxArray_int32_T rows,
                                       const int32_T b_rows,
                                       emxArray_creal_T newIm);

static __global__ void conv2_kernel141(const real_T cv_im, const real_T cv_re,
                                       const int32_T offsetH,
                                       emxArray_creal_T c);

static __global__ void conv2_kernel43(const emxArray_real_T a,
                                      const int32_T b_a, emxArray_real_T c_a,
                                      int32_T a_dim0);

static __global__ void conv2_kernel44(const int32_T a, emxArray_real_T c);

static __global__ void conv2_kernel45(const real_T b, const emxArray_real_T a,
                                      emxArray_real_T c);

static __global__ void conv2_kernel46(const emxArray_real_T c,
                                      const int32_T b_c, emxArray_real_T c_c,
                                      int32_T c_dim0);

static __global__ void conv2_kernel47(const int32_T a,
                                      emxArray_real_T expanded);

static __global__ void conv2_kernel48(const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel49(const emxArray_real_T a,
                                      const emxArray_int32_T iv,
                                      const int32_T b,
                                      emxArray_real_T expanded);

static __global__ void conv2_kernel50(const real_T b,
                                      const emxArray_real_T expanded,
                                      const int32_T c, emxArray_real_T b_c);

static __global__ void conv2_kernel51(const int32_T b, emxArray_real_T c);

static __global__ void conv2_kernel52(const int32_T b, emxArray_real_T c);

static __global__ void conv2_kernel53(const emxArray_real_T a,
                                      const emxArray_real_T b,
                                      emxArray_real_T c);

static __global__ void conv2_kernel54(const int32_T OH,
                                      emxArray_real_T expanded);

static __global__ void conv2_kernel55(const int32_T offsetH,
                                      const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv1);

static __global__ void conv2_kernel56(const emxArray_real_T b, const int32_T mc,
                                      const emxArray_int32_T iv1,
                                      const int32_T c, emxArray_real_T expanded,
                                      int32_T expanded_dim0);

static __global__ void conv2_kernel57(const emxArray_real_T expanded,
                                      const emxArray_int32_T cols,
                                      const emxArray_int32_T rows,
                                      const emxArray_real_T a, const int32_T b,
                                      const int32_T c, emxArray_real_T b_c,
                                      int32_T a_dim0, int32_T a_dim1,
                                      int32_T expanded_dim0, int32_T c_dim0);

// Function Definitions
//
//
namespace coder {
static void
b_conv2NonSeparable(emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
                    emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
                    emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
                    emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
                    emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *cpu_expanded;
  int32_T k;
  int32_T mc;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#b_conv2NonSeparable#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_expanded);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  k = cpu_c->size[0];
  cpu_c->size[0] = mc;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_c, k, &ag_emlrtRTEI);
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block, 2147483647U);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#conv2NonSeparable_kernel66#" MW_AT_LINE);
    conv2NonSeparable_kernel66<<<grid, block>>>(mc - 1, *gpu_c);
  }
  *c_outdatedOnGpu = false;
  *c_outdatedOnCpu = true;
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_expanded, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  if ((cpu_a->size[0] != 0) && (cpu_b->size[0] != 0)) {
    if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
      real_T blockDims_idx_0;
      if (*b_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      blockDims_idx_0 = cpu_b->data[0];
      k = cpu_c->size[0];
      cpu_c->size[0] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, k, &ag_emlrtRTEI);
      if (*a_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
      }
      *a_outdatedOnCpu = false;
      cpu_c->data[0] = cpu_a->data[0] * blockDims_idx_0;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      real_T blockDims_idx_0;
      int32_T b_OH;
      int32_T n;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims_idx_0;
      OH = (static_cast<uint32_T>(cpu_a->size[0]) +
            static_cast<uint32_T>(cpu_b->size[0])) -
           1U;
      mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      blockDims_idx_0 = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
          static_cast<real_T>(mc));
      threadDims_idx_0 = static_cast<int8_T>(mc);
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
      if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_b->size[0];
      }
      k = cpu_expanded->size[0];
      cpu_expanded->size[0] = mc - 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_expanded, k, &cg_emlrtRTEI);
      mc = (static_cast<int32_T>(OH) + cpu_b->size[0]) - 2;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_expanded, &gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2NonSeparable_kernel67#" MW_AT_LINE);
        conv2NonSeparable_kernel67<<<grid, block>>>(mc, gpu_expanded);
      }
      n = cpu_a->size[0];
      k = cpu_y->size[0] * cpu_y->size[1];
      cpu_y->size[0] = 1;
      cpu_y->size[1] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_y, k, &dg_emlrtRTEI);
      cpu_y->data[0] = 1;
      mc = 1;
      profileLoopStart("b_conv2NonSeparable_loop_0", __LINE__, (n - 2) + 1, "");
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_y->data[k + 1] = mc;
      }
      profileLoopEnd();
      k = cpu_iv->size[0];
      cpu_iv->size[0] = cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_iv, k, &cg_emlrtRTEI);
      mc = cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2NonSeparable_kernel68#" MW_AT_LINE);
        conv2NonSeparable_kernel68<<<grid, block>>>(offsetH, gpu_y, mc, gpu_iv);
      }
      mc = cpu_iv->size[0];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
      if (*a_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
      }
      *a_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2NonSeparable_kernel69#" MW_AT_LINE);
        conv2NonSeparable_kernel69<<<grid, block>>>(*gpu_a, gpu_iv, mc - 1,
                                                    gpu_expanded);
      }
      n = cpu_b->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_rows, k, &dg_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      profileLoopStart("b_conv2NonSeparable_loop_1", __LINE__, (n - 2) + 1, "");
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      profileLoopEnd();
      k = cpu_c->size[0];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, k, &cg_emlrtRTEI);
      if (blockDims_idx_0 < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims_idx_0);
      } else {
        OH = MAX_uint32_T;
      }
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwApplyLaunchParameters(
          computeNumIters(b_OH - 1), dim3(OH, 1U, 1U),
          dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid, &block);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      if (*b_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2NonSeparable_kernel70#" MW_AT_LINE);
        conv2NonSeparable_kernel70<<<grid, block>>>(
            gpu_expanded, gpu_rows, *gpu_b, b_OH - 1, *gpu_c, cpu_b->size[0U]);
      }
    }
  }
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxRangePop();
}

//
//
static void
c_conv2NonSeparable(emxArray_creal_T *cpu_a, emxArray_creal_T *gpu_a,
                    boolean_T *a_outdatedOnGpu, const emxArray_real_T *b,
                    emxArray_creal_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_creal_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_a;
  emxArray_creal_T gpu_expanded;
  emxArray_creal_T gpu_newIm;
  emxArray_creal_T *b_cpu_a;
  emxArray_creal_T *cpu_expanded;
  emxArray_creal_T *cpu_newIm;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  creal_T(*gpu_b_data)[1];
  int32_T i;
  int32_T mc;
  int32_T nc;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#c_conv2NonSeparable#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_b_data, 16UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_expanded);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (b->size[0] == 0)) {
    mc = cpu_a->size[0] + b->size[0];
  } else {
    mc = (cpu_a->size[0] + b->size[0]) - 1;
  }
  if (cpu_a->size[1] == 0) {
    nc = 1;
  } else {
    nc = cpu_a->size[1];
  }
  i = cpu_c->size[0] * cpu_c->size[1];
  cpu_c->size[0] = mc;
  cpu_c->size[1] = nc;
  nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
  emxEnsureCapacity_creal_T(cpu_c, i, &ag_emlrtRTEI);
  i = mc * nc - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#conv2NonSeparable_kernel119#" MW_AT_LINE);
    conv2NonSeparable_kernel119<<<grid, block>>>(i, *gpu_c);
  }
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_expanded, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_newIm, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_a, 1, &ig_emlrtRTEI, true);
  if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0) || (b->size[0] == 0)) {
    i = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, i, &ag_emlrtRTEI);
    i = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel120#" MW_AT_LINE);
      conv2NonSeparable_kernel120<<<grid, block>>>(i, *gpu_c);
    }
  } else if ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1) &&
             (b->size[0] == 1)) {
    real_T cv_re;
    cv_re = b->data[0];
    nvtxMarkA("#conv2NonSeparable_kernel121#" MW_AT_LINE);
    conv2NonSeparable_kernel121<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        cv_re, *gpu_b_data);
    i = b_cpu_a->size[0];
    b_cpu_a->size[0] = 1;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(b_cpu_a, i, &ig_emlrtRTEI);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(b_cpu_a, &b_gpu_a, true);
    if (*a_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    nvtxMarkA("#conv2NonSeparable_kernel122#" MW_AT_LINE);
    conv2NonSeparable_kernel122<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_a, *gpu_b_data, b_gpu_a);
    i = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, i, &ag_emlrtRTEI);
    i = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel123#" MW_AT_LINE);
      conv2NonSeparable_kernel123<<<grid, block>>>(b_gpu_a, i, *gpu_c);
    }
  } else {
    int32_T iv[2];
    int32_T OW;
    int32_T b_OH;
    int32_T offsetH;
    uint32_T OH;
    boolean_T c_needsGpuEnsureCapacity;
    boolean_T rows_needsGpuEnsureCapacity;
    boolean_T rows_outdatedOnGpu;
    OH = (static_cast<uint32_T>(cpu_a->size[0]) +
          static_cast<uint32_T>(b->size[0])) -
         1U;
    OW = cpu_a->size[1];
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    b_OH = static_cast<int32_T>(OH);
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(b->size[0]) / 2.0)) +
              static_cast<int32_T>(
                  std::floor((static_cast<real_T>(b->size[0]) - 1.0) / 2.0));
    if (b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
      mc = MAX_int32_T;
    } else {
      mc = static_cast<int32_T>(OH) + b->size[0];
    }
    i = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[0] = mc - 1;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_expanded, i, &cg_emlrtRTEI);
    if (cpu_a->size[1] > 2147483646) {
      mc = MAX_int32_T;
    } else {
      mc = cpu_a->size[1] + 1;
    }
    i = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[1] = mc - 1;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_expanded, i, &cg_emlrtRTEI);
    i = ((static_cast<int32_T>(OH) + b->size[0]) - 1) * cpu_a->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel124#" MW_AT_LINE);
      conv2NonSeparable_kernel124<<<grid, block>>>(i, gpu_expanded);
    }
    nc = cpu_a->size[0];
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_a->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_y, i, &dg_emlrtRTEI);
    cpu_y->data[0] = 1;
    mc = 1;
    profileLoopStart("c_conv2NonSeparable_loop_0", __LINE__, (nc - 2) + 1, "");
    for (int32_T k{0}; k <= nc - 2; k++) {
      mc++;
      cpu_y->data[k + 1] = mc;
    }
    profileLoopEnd();
    i = cpu_iv->size[0];
    cpu_iv->size[0] = cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_iv, i, &cg_emlrtRTEI);
    i = cpu_y->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel125#" MW_AT_LINE);
      conv2NonSeparable_kernel125<<<grid, block>>>(offsetH, gpu_y, i, gpu_iv);
    }
    nc = cpu_a->size[1];
    i = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_a->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(b_cpu_y, i, &dg_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    mc = 1;
    profileLoopStart("c_conv2NonSeparable_loop_1", __LINE__, (nc - 2) + 1, "");
    for (int32_T k{0}; k <= nc - 2; k++) {
      mc++;
      b_cpu_y->data[k + 1] = mc;
    }
    profileLoopEnd();
    i = cpu_iv1->size[0];
    cpu_iv1->size[0] = b_cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_iv1, i, &cg_emlrtRTEI);
    i = b_cpu_y->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel126#" MW_AT_LINE);
      conv2NonSeparable_kernel126<<<grid, block>>>(b_gpu_y, i, gpu_iv1);
    }
    iv[0] = cpu_iv->size[0];
    iv[1] = cpu_iv1->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(iv[1] - 1, iv[0] - 1), &grid,
                            &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    if (*a_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel127#" MW_AT_LINE);
      conv2NonSeparable_kernel127<<<grid, block>>>(
          *gpu_a, iv[0], gpu_iv1, gpu_iv, iv[0] - 1, iv[1] - 1, gpu_expanded,
          cpu_expanded->size[0U]);
    }
    nc = b->size[0];
    i = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = b->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_rows, i, &dg_emlrtRTEI);
    cpu_rows->data[0] = 0;
    rows_outdatedOnGpu = true;
    rows_needsGpuEnsureCapacity = true;
    mc = 0;
    profileLoopStart("c_conv2NonSeparable_loop_2", __LINE__, (nc - 2) + 1, "");
    for (int32_T k{0}; k <= nc - 2; k++) {
      mc++;
      cpu_rows->data[k + 1] = mc;
    }
    profileLoopEnd();
    i = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = static_cast<int32_T>(OH);
    cpu_c->size[1] = cpu_a->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, i, &cg_emlrtRTEI);
    c_needsGpuEnsureCapacity = true;
    mc = b->size[0];
    profileLoopStart("c_conv2NonSeparable_loop_3", __LINE__, (OW - 1) + 1, "");
    for (offsetH = 0; offsetH < OW; offsetH++) {
      profileLoopStart("c_conv2NonSeparable_loop_4", __LINE__, (b_OH - 1) + 1,
                       "");
      for (int32_T k{0}; k < b_OH; k++) {
        real_T cv_im;
        real_T cv_re;
        boolean_T newIm_outdatedOnCpu;
        i = cpu_newIm->size[0];
        cpu_newIm->size[0] = cpu_rows->size[1];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_newIm, i, &cg_emlrtRTEI);
        i = cpu_rows->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        if (rows_needsGpuEnsureCapacity) {
          nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows,
                                       !rows_outdatedOnGpu);
        }
        rows_needsGpuEnsureCapacity = false;
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_newIm, &gpu_newIm, true);
        if (rows_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
        }
        rows_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2NonSeparable_kernel128#" MW_AT_LINE);
          conv2NonSeparable_kernel128<<<grid, block>>>(offsetH, gpu_expanded, k,
                                                       gpu_rows, i, gpu_newIm,
                                                       cpu_expanded->size[0U]);
        }
        newIm_outdatedOnCpu = true;
        cv_re = 0.0;
        cv_im = 0.0;
        profileLoopStart("c_conv2NonSeparable_loop_5", __LINE__, (mc - 1) + 1,
                         "");
        for (nc = 0; nc < mc; nc++) {
          if (newIm_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_creal_T(cpu_newIm, &gpu_newIm);
          }
          cv_re += cpu_newIm->data[nc].re * b->data[(b->size[0] - nc) - 1];
          newIm_outdatedOnCpu = false;
          cv_im += cpu_newIm->data[nc].im * b->data[(b->size[0] - nc) - 1];
        }
        profileLoopEnd();
        if (c_needsGpuEnsureCapacity) {
          nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
        }
        nvtxMarkA("#conv2NonSeparable_kernel129#" MW_AT_LINE);
        conv2NonSeparable_kernel129<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            cv_re, offsetH, k, *gpu_c, cpu_c->size[0U]);
        c_needsGpuEnsureCapacity = false;
        nvtxMarkA("#conv2NonSeparable_kernel130#" MW_AT_LINE);
        conv2NonSeparable_kernel130<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            cv_im, offsetH, k, *gpu_c, cpu_c->size[0U]);
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_a);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_newIm);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_a);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_b_data), __FILE__, __LINE__);
  *c_outdatedOnCpu = true;
  *c_outdatedOnGpu = false;
  nvtxRangePop();
}

//
//
static void
conv2NonSeparable(emxArray_real_T *cpu_a, emxArray_real_T *gpu_a,
                  boolean_T *a_outdatedOnGpu, emxArray_real_T *cpu_b,
                  boolean_T *b_outdatedOnCpu, emxArray_real_T *gpu_b,
                  boolean_T *b_outdatedOnGpu, emxArray_real_T *cpu_c,
                  boolean_T *c_outdatedOnCpu, emxArray_real_T *gpu_c,
                  boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T b_gpu_c;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *cpu_expanded;
  int32_T k;
  int32_T mc;
  int32_T nc;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#conv2NonSeparable#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_c);
  *c_outdatedOnCpu = false;
  *c_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  if (cpu_a->size[1] == 0) {
    nc = 1;
  } else {
    nc = cpu_a->size[1];
  }
  k = cpu_c->size[0] * cpu_c->size[1];
  cpu_c->size[0] = mc;
  cpu_c->size[1] = nc;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_c, k, &ag_emlrtRTEI);
  k = mc * nc - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#conv2NonSeparable_kernel58#" MW_AT_LINE);
    conv2NonSeparable_kernel58<<<grid, block>>>(k, *gpu_c);
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_c, 1, &gg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_expanded, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0) || (cpu_b->size[0] == 0)) {
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_c, k, &ag_emlrtRTEI);
    k = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel59#" MW_AT_LINE);
      conv2NonSeparable_kernel59<<<grid, block>>>(k, *gpu_c);
    }
  } else if ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1) &&
             (cpu_b->size[0] == 1)) {
    if (*b_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
    }
    *b_outdatedOnCpu = false;
    nvtxMarkA("#mtimes#" MW_AT_LINE);
    internal::blas::mtimes(cpu_a, gpu_a, a_outdatedOnGpu,
                           (const real_T *)cpu_b->data, b_cpu_c,
                           c_outdatedOnCpu, &b_gpu_c, c_outdatedOnGpu);
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_c, k, &ag_emlrtRTEI);
    k = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, !*c_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    if (*c_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_c, b_cpu_c);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel60#" MW_AT_LINE);
      conv2NonSeparable_kernel60<<<grid, block>>>(b_gpu_c, k, *gpu_c);
    }
  } else {
    real_T blockDims[3];
    int32_T iv[2];
    int32_T OW;
    int32_T b_OH;
    int32_T offsetH;
    uint32_T OH;
    int8_T threadDims[3];
    OH = (static_cast<uint32_T>(cpu_a->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    nc = static_cast<int32_T>(
        std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
    blockDims[0] =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
                   static_cast<real_T>(mc));
    blockDims[1] =
        std::floor((static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                                        static_cast<uint32_T>(nc)) -
                    1.0) /
                   static_cast<real_T>(nc));
    threadDims[0] = static_cast<int8_T>(mc);
    threadDims[1] = static_cast<int8_T>(nc);
    OW = cpu_a->size[1];
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    b_OH = static_cast<int32_T>(OH);
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
      mc = MAX_int32_T;
    } else {
      mc = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[0] = mc - 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_expanded, k, &cg_emlrtRTEI);
    if (cpu_a->size[1] > 2147483646) {
      mc = MAX_int32_T;
    } else {
      mc = cpu_a->size[1] + 1;
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[1] = mc - 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_expanded, k, &cg_emlrtRTEI);
    k = ((static_cast<int32_T>(OH) + cpu_b->size[0]) - 1) * cpu_a->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel61#" MW_AT_LINE);
      conv2NonSeparable_kernel61<<<grid, block>>>(k, gpu_expanded);
    }
    nc = cpu_a->size[0];
    k = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_a->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_y, k, &dg_emlrtRTEI);
    cpu_y->data[0] = 1;
    mc = 1;
    profileLoopStart("conv2NonSeparable_loop_0", __LINE__, (nc - 2) + 1, "");
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_y->data[k + 1] = mc;
    }
    profileLoopEnd();
    k = cpu_iv->size[0];
    cpu_iv->size[0] = cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_iv, k, &cg_emlrtRTEI);
    k = cpu_y->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel62#" MW_AT_LINE);
      conv2NonSeparable_kernel62<<<grid, block>>>(offsetH, gpu_y, k, gpu_iv);
    }
    nc = cpu_a->size[1];
    k = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_a->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(b_cpu_y, k, &dg_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    mc = 1;
    profileLoopStart("conv2NonSeparable_loop_1", __LINE__, (nc - 2) + 1, "");
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      b_cpu_y->data[k + 1] = mc;
    }
    profileLoopEnd();
    k = cpu_iv1->size[0];
    cpu_iv1->size[0] = b_cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_iv1, k, &cg_emlrtRTEI);
    k = b_cpu_y->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel63#" MW_AT_LINE);
      conv2NonSeparable_kernel63<<<grid, block>>>(b_gpu_y, k, gpu_iv1);
    }
    iv[0] = cpu_iv->size[0];
    iv[1] = cpu_iv1->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(iv[1] - 1, iv[0] - 1), &grid,
                            &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    if (*a_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel64#" MW_AT_LINE);
      conv2NonSeparable_kernel64<<<grid, block>>>(
          *gpu_a, iv[0], gpu_iv1, gpu_iv, iv[0] - 1, iv[1] - 1, gpu_expanded,
          cpu_expanded->size[0U]);
    }
    nc = cpu_b->size[0];
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = cpu_b->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_rows, k, &dg_emlrtRTEI);
    cpu_rows->data[0] = 0;
    mc = 0;
    profileLoopStart("conv2NonSeparable_loop_2", __LINE__, (nc - 2) + 1, "");
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_rows->data[k + 1] = mc;
    }
    profileLoopEnd();
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = static_cast<int32_T>(OH);
    cpu_c->size[1] = cpu_a->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_c, k, &cg_emlrtRTEI);
    if (blockDims[0] < 4.294967296E+9) {
      OH = static_cast<uint32_T>(blockDims[0]);
    } else {
      OH = MAX_uint32_T;
    }
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                            dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                            dim3(static_cast<uint32_T>(threadDims[0]),
                                 static_cast<uint32_T>(threadDims[1]), 1U),
                            &grid, &block);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel65#" MW_AT_LINE);
      conv2NonSeparable_kernel65<<<grid, block>>>(
          gpu_expanded, gpu_rows, *gpu_b, b_OH - 1, OW - 1, *gpu_c,
          cpu_b->size[0U], cpu_expanded->size[0U], cpu_c->size[0U]);
    }
  }
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_expanded);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  *c_outdatedOnCpu = true;
  *c_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace coder
static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel119(
    const int32_T b, emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1].re = 0.0;
    c.data[i1].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel120(
    const int32_T b, emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1].re = 0.0;
    c.data[i1].im = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel121(
    const real_T cv_re, creal_T b_data[1])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0].im = 0.0;
    b_data[0].re = cv_re;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel122(
    const emxArray_creal_T a, const creal_T b_data[1], emxArray_creal_T b_a)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 1) {
    real_T d;
    b_a.data[0].re = 0.0;
    b_a.data[0].im = 0.0;
    d = b_data[0].re;
    b_a.data[0].re += a.data[0].re * d - a.data[0].im * 0.0;
    b_a.data[0].im += a.data[0].re * 0.0 + a.data[0].im * d;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel123(
    const emxArray_creal_T a, const int32_T iv, emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(iv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1] = a.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel124(
    const int32_T OH, emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    expanded.data[i1].re = 0.0;
    expanded.data[i1].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel125(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    i1 = static_cast<int32_T>(idx);
    q1 = y.data[i1];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[i1] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel126(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    iv1.data[i1] = y.data[i1] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel127(
    const emxArray_creal_T a, const int32_T iv_dim0, const emxArray_int32_T iv1,
    const emxArray_int32_T iv, const int32_T b_iv, const int32_T c_iv,
    emxArray_creal_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_iv) + 1UL) *
                (static_cast<uint64_T>(b_iv) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    q1 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_iv) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(q1)) /
                              (static_cast<uint64_T>(b_iv) + 1UL));
    expanded.data[iv.data[q1] + expanded_dim0 * iv1.data[i1]] =
        a.data[q1 + iv_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel128(
    const int32_T offsetH, const emxArray_creal_T expanded, const int32_T k,
    const emxArray_int32_T rows, const int32_T b_rows, emxArray_creal_T newIm,
    int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_rows);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    i1 = static_cast<int32_T>(idx);
    q1 = rows.data[i1];
    if (q1 > 2147483646 - k) {
      q1 = MAX_int32_T;
    } else {
      q1 = (k + q1) + 1;
    }
    newIm.data[i1] = expanded.data[(q1 + expanded_dim0 * offsetH) - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel129(
    const real_T cv_re, const int32_T offsetH, const int32_T k,
    emxArray_creal_T c, int32_T c_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[k + c_dim0 * offsetH].re = cv_re;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel130(
    const real_T cv_im, const int32_T offsetH, const int32_T k,
    emxArray_creal_T c, int32_T c_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[k + c_dim0 * offsetH].im = cv_im;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel58(
    const int32_T b, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel59(
    const int32_T b, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel60(
    const emxArray_real_T c, const int32_T b, emxArray_real_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_c.data[i] = c.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel61(
    const int32_T OH, emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    expanded.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel62(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T q1;
    i = static_cast<int32_T>(idx);
    q1 = y.data[i];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[i] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel63(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    iv1.data[i] = y.data[i] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel64(
    const emxArray_real_T a, const int32_T iv_dim0, const emxArray_int32_T iv1,
    const emxArray_int32_T iv, const int32_T b_iv, const int32_T c_iv,
    emxArray_real_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_iv) + 1UL) *
                (static_cast<uint64_T>(b_iv) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T ocol;
    ocol = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_iv) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(ocol)) /
                             (static_cast<uint64_T>(b_iv) + 1UL));
    expanded.data[iv.data[ocol] + expanded_dim0 * iv1.data[i]] =
        a.data[ocol + iv_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel65(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_real_T b, const int32_T c, const int32_T d,
    emxArray_real_T b_c, int32_T b_dim0, int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(d) + 1UL) * (static_cast<uint64_T>(c) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T ocol;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(c) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(c) + 1UL));
    b_cv = 0.0;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[(q1 + expanded_dim0 * ocol) - 1] *
              b.data[(b_dim0 - m) - 1];
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel66(
    const int32_T b, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel67(
    const int32_T OH, emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel68(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel69(
    const emxArray_real_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv.data[orow]] = a.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel70(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_real_T b, const int32_T c, emxArray_real_T b_c,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_cv = 0.0;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[q1 - 1] * b.data[(b_dim0 - m) - 1];
    }
    b_c.data[orow] = b_cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel102(
    const emxArray_creal_T a, const int32_T b_a, emxArray_creal_T c_a,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c_a.data[qY] = a.data[a_dim0 * qY];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel103(const int32_T a,
                                                            emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0;
    c.data[qY].im = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel104(
    const emxArray_creal_T a, const real_T b, real_T *d1, real_T *d)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *d = b * a.data[0].re;
    *d1 = b * a.data[0].im;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void conv2_kernel105(const real_T *d,
                                                            const real_T *d1,
                                                            emxArray_creal_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0].im = *d1;
    c.data[0].re = *d;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel106(
    const emxArray_creal_T c, const int32_T b_c, emxArray_creal_T c_c,
    int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c_c.data[c_dim0 * qY] = c.data[qY];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel107(const int32_T a,
                                                    emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[qY].re = 0.0;
    expanded.data[qY].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel108(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    iv.data[qY] = y.data[qY] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel109(
    const emxArray_creal_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[iv.data[qY]] = a.data[qY];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel110(
    const emxArray_creal_T expanded, const real_T b, const int32_T c,
    emxArray_creal_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_c.data[orow].re = b * expanded.data[orow].re;
    b_c.data[orow].im = b * expanded.data[orow].im;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel111(const int32_T b,
                                                            emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0;
    c.data[qY].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel112(const int32_T b,
                                                            emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0;
    c.data[qY].im = 0.0;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void conv2_kernel113(const real_T d,
                                                            creal_T b_data[1])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0].im = 0.0;
    b_data[0].re = d;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel114(
    const emxArray_creal_T a, const creal_T b_data[1], emxArray_creal_T c)
{
  int32_T k;
  k = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (k < 1) {
    creal_T b;
    b.re = b_data[0].re * a.data[0].re - 0.0 * a.data[0].im;
    b.im = b_data[0].re * a.data[0].im + 0.0 * a.data[0].re;
    c.data[0] = b;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel115(const int32_T OH,
                                                    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[qY] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel116(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T q1;
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    q1 = y.data[qY];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[qY] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel117(
    const emxArray_real_T b, const int32_T mc, const emxArray_int32_T iv1,
    const int32_T c, emxArray_real_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[iv1.data[qY] + expanded_dim0 * mc] = b.data[qY];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel118(
    const emxArray_real_T expanded, const emxArray_int32_T cols,
    const emxArray_int32_T rows, const emxArray_creal_T a, const int32_T b,
    const int32_T c, emxArray_creal_T b_c, int32_T a_dim0, int32_T a_dim1,
    int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    creal_T b_cv;
    int32_T ocol;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_cv.re = 0.0;
    b_cv.im = 0.0;
    for (int32_T n{0}; n < a_dim1; n++) {
      for (int32_T m{0}; m < a_dim0; m++) {
        creal_T b_b;
        int32_T q1;
        int32_T qY;
        b_b = a.data[((a_dim0 - m) + a_dim0 * ((a_dim1 - n) - 1)) - 1];
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv.re += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] * b_b.re;
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv.im += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] * b_b.im;
      }
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel131(const int32_T b,
                                                            emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow].re = 0.0;
    c.data[orow].im = 0.0;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel132(const int32_T OH,
                                                    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel133(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel134(
    const emxArray_real_T b, const emxArray_int32_T iv1, const int32_T c,
    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv1.data[orow]] = b.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel135(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_creal_T a, const int32_T b, emxArray_creal_T c,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    creal_T b_cv;
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_cv.re = 0.0;
    b_cv.im = 0.0;
    for (int32_T m{0}; m < a_dim0; m++) {
      creal_T b_b;
      int32_T q1;
      b_b = a.data[(a_dim0 - m) - 1];
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv.re += expanded.data[q1 - 1] * b_b.re;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv.im += expanded.data[q1 - 1] * b_b.im;
    }
    c.data[orow] = b_cv;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel136(const int32_T b,
                                                            emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow].re = 0.0;
    c.data[orow].im = 0.0;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel137(const int32_T OH,
                                                    emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow].re = 0.0;
    expanded.data[orow].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel138(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel139(
    const emxArray_creal_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv.data[orow]] = a.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel140(
    const emxArray_creal_T expanded, const int32_T offsetH,
    const emxArray_int32_T rows, const int32_T b_rows, emxArray_creal_T newIm)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_rows);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = rows.data[orow];
    if (q1 > 2147483646 - offsetH) {
      q1 = MAX_int32_T;
    } else {
      q1 = (offsetH + q1) + 1;
    }
    newIm.data[orow] = expanded.data[q1 - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel141(
    const real_T cv_im, const real_T cv_re, const int32_T offsetH,
    emxArray_creal_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[offsetH].re = cv_re;
    c.data[offsetH].im = cv_im;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel43(
    const emxArray_real_T a, const int32_T b_a, emxArray_real_T c_a,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c_a.data[ocol] = a.data[a_dim0 * ocol];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel44(const int32_T a,
                                                           emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel45(
    const real_T b, const emxArray_real_T a, emxArray_real_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = a.data[0] * b;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel46(
    const emxArray_real_T c, const int32_T b_c, emxArray_real_T c_c,
    int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c_c.data[c_dim0 * ocol] = c.data[ocol];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel47(const int32_T a,
                                                   emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel48(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    iv.data[ocol] = y.data[ocol] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel49(
    const emxArray_real_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[iv.data[ocol]] = a.data[ocol];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel50(
    const real_T b, const emxArray_real_T expanded, const int32_T c,
    emxArray_real_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_c.data[orow] = expanded.data[orow] * b;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel51(const int32_T b,
                                                           emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel52(const int32_T b,
                                                           emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel53(
    const emxArray_real_T a, const emxArray_real_T b, emxArray_real_T c)
{
  int32_T k;
  k = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (k < 1) {
    real_T b_cv;
    b_cv = b.data[0] * a.data[0];
    c.data[0] = b_cv;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel54(const int32_T OH,
                                                   emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel55(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    int32_T q1;
    ocol = static_cast<int32_T>(idx);
    q1 = y.data[ocol];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[ocol] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel56(
    const emxArray_real_T b, const int32_T mc, const emxArray_int32_T iv1,
    const int32_T c, emxArray_real_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[iv1.data[ocol] + expanded_dim0 * mc] = b.data[ocol];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel57(
    const emxArray_real_T expanded, const emxArray_int32_T cols,
    const emxArray_int32_T rows, const emxArray_real_T a, const int32_T b,
    const int32_T c, emxArray_real_T b_c, int32_T a_dim0, int32_T a_dim1,
    int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T ocol;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_cv = 0.0;
    for (int32_T n{0}; n < a_dim1; n++) {
      for (int32_T m{0}; m < a_dim0; m++) {
        int32_T q1;
        int32_T qY;
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] *
                a.data[((a_dim0 - m) + a_dim0 * ((a_dim1 - n) - 1)) - 1];
      }
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

//
//
namespace coder {
void b_conv2(const emxArray_real_T *cpu_a, emxArray_real_T *cpu_b,
             boolean_T *b_outdatedOnCpu, emxArray_real_T *gpu_b,
             boolean_T *b_outdatedOnGpu, emxArray_real_T *cpu_c,
             boolean_T *c_outdatedOnCpu, emxArray_real_T *gpu_c,
             boolean_T *c_outdatedOnGpu)
{
  emxArray_real_T gpu_a;
  boolean_T a_outdatedOnCpu;
  boolean_T a_outdatedOnGpu;
  nvtxRangePushA("#fcn#b_conv2#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_a);
  a_outdatedOnCpu = false;
  a_outdatedOnGpu = true;
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    int32_T i;
    i = cpu_c->size[0];
    cpu_c->size[0] = 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_c, i, &bg_emlrtRTEI);
    if (*b_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
    }
    *b_outdatedOnCpu = false;
    cpu_c->data[0] = cpu_a->data[0] * cpu_b->data[0];
    *c_outdatedOnCpu = false;
    *c_outdatedOnGpu = true;
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    nvtxMarkA("#b_conv2NonSeparable#" MW_AT_LINE);
    b_conv2NonSeparable(cpu_b, b_outdatedOnCpu, gpu_b, b_outdatedOnGpu,
                        (emxArray_real_T *)cpu_a, &a_outdatedOnCpu, &gpu_a,
                        &a_outdatedOnGpu, cpu_c, c_outdatedOnCpu, gpu_c,
                        c_outdatedOnGpu);
  } else {
    nvtxMarkA("#b_conv2NonSeparable#" MW_AT_LINE);
    b_conv2NonSeparable((emxArray_real_T *)cpu_a, &a_outdatedOnCpu, &gpu_a,
                        &a_outdatedOnGpu, cpu_b, b_outdatedOnCpu, gpu_b,
                        b_outdatedOnGpu, cpu_c, c_outdatedOnCpu, gpu_c,
                        c_outdatedOnGpu);
  }
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_a);
  nvtxRangePop();
}

//
//
void c_conv2(const emxArray_creal_T *cpu_a, emxArray_real_T *cpu_b,
             boolean_T *b_outdatedOnCpu, emxArray_real_T *gpu_b,
             boolean_T *b_outdatedOnGpu, emxArray_creal_T *cpu_c,
             boolean_T *c_outdatedOnCpu, emxArray_creal_T *gpu_c,
             boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_a;
  emxArray_creal_T b_gpu_c;
  emxArray_creal_T gpu_a;
  emxArray_creal_T gpu_expanded;
  emxArray_creal_T *b_cpu_a;
  emxArray_creal_T *b_cpu_c;
  emxArray_creal_T *cpu_expanded;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_cols;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_cols;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T b_gpu_expanded;
  emxArray_real_T *b_cpu_expanded;
  creal_T(*gpu_b_data)[1];
  real_T *gpu_d;
  real_T *gpu_d1;
  boolean_T a_outdatedOnGpu;
  nvtxRangePushA("#fcn#c_conv2#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_d1, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_d, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_b_data, 16UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_cols);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_a);
  a_outdatedOnGpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_c, 1, &eg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_a, 1, &wf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_expanded, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_expanded, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_cols, 2, &cg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    if (cpu_a->size[1] < 1) {
      int32_T k;
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_c, k, &yf_emlrtRTEI);
      cpu_c->data[0].im = 0.0;
      cpu_c->data[0].re = 0.0;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      real_T cpu_d;
      int32_T k;
      boolean_T validLaunchParams;
      k = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(b_cpu_a, k, &wf_emlrtRTEI);
      k = cpu_a->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(b_cpu_a, &b_gpu_a, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel102#" MW_AT_LINE);
        conv2_kernel102<<<grid, block>>>(gpu_a, k, b_gpu_a, cpu_a->size[0U]);
      }
      if (*b_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      cpu_d = cpu_b->data[0];
      k = b_cpu_c->size[0];
      b_cpu_c->size[0] = b_cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(b_cpu_c, k, &ag_emlrtRTEI);
      k = b_cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(b_cpu_c, &b_gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel103#" MW_AT_LINE);
        conv2_kernel103<<<grid, block>>>(k, b_gpu_c);
      }
      if (b_cpu_a->size[0] == 1) {
        cpu_d = cpu_b->data[0];
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(b_cpu_c, k, &ag_emlrtRTEI);
        nvtxMarkA("#conv2_kernel104#" MW_AT_LINE);
        conv2_kernel104<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b_gpu_a, cpu_d,
                                                                 gpu_d1, gpu_d);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(b_cpu_c, &b_gpu_c, true);
        nvtxMarkA("#conv2_kernel105#" MW_AT_LINE);
        conv2_kernel105<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_d, gpu_d1,
                                                                 b_gpu_c);
      } else {
        int32_T b_OH;
        int32_T mc;
        int32_T nc;
        int32_T offsetH;
        int8_T threadDims[3];
        nc = static_cast<int32_T>(std::fmin(
            32.0, (static_cast<real_T>(b_cpu_a->size[0]) + 1.0) - 1.0));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(b_cpu_a->size[0]) +
                                 static_cast<uint32_T>(nc)) -
             1.0) /
            static_cast<real_T>(nc)));
        threadDims[0] = static_cast<int8_T>(nc);
        b_OH = b_cpu_a->size[0];
        if (b_cpu_a->size[0] > 2147483646) {
          mc = MAX_int32_T;
        } else {
          mc = b_cpu_a->size[0] + 1;
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_expanded, k, &cg_emlrtRTEI);
        k = b_cpu_a->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel107#" MW_AT_LINE);
          conv2_kernel107<<<grid, block>>>(k, gpu_expanded);
        }
        nc = b_cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = b_cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_y, k, &dg_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        profileLoopStart("c_conv2_loop_0", __LINE__, (nc - 2) + 1, "");
        for (k = 0; k <= nc - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, k, &cg_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel108#" MW_AT_LINE);
          conv2_kernel108<<<grid, block>>>(gpu_y, k, gpu_iv);
        }
        nc = cpu_iv->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel109#" MW_AT_LINE);
          conv2_kernel109<<<grid, block>>>(b_gpu_a, gpu_iv, nc - 1,
                                           gpu_expanded);
        }
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = b_cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(b_cpu_c, k, &cg_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims[0]), 1U, 1U), &grid, &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(b_cpu_c, &b_gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel110#" MW_AT_LINE);
          conv2_kernel110<<<grid, block>>>(gpu_expanded, cpu_d, b_OH - 1,
                                           b_gpu_c);
        }
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = b_cpu_c->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_c, k, &bg_emlrtRTEI);
      k = b_cpu_c->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel106#" MW_AT_LINE);
        conv2_kernel106<<<grid, block>>>(b_gpu_c, k, *gpu_c, cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    int32_T nc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    if (cpu_a->size[1] == 0) {
      nc = 1;
    } else {
      nc = cpu_a->size[1];
    }
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, k, &xf_emlrtRTEI);
    k = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2_kernel111#" MW_AT_LINE);
      conv2_kernel111<<<grid, block>>>(k, *gpu_c);
    }
    if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0)) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = nc;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_c, k, &xf_emlrtRTEI);
      k = mc * nc - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel112#" MW_AT_LINE);
        conv2_kernel112<<<grid, block>>>(k, *gpu_c);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else if ((cpu_b->size[0] == 1) &&
               ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1))) {
      real_T cpu_d;
      if (*b_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      cpu_d = cpu_b->data[0];
      nvtxMarkA("#conv2_kernel113#" MW_AT_LINE);
      conv2_kernel113<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(cpu_d,
                                                               *gpu_b_data);
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_c, k, &xf_emlrtRTEI);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_a, cpu_a);
      nvtxMarkA("#conv2_kernel114#" MW_AT_LINE);
      conv2_kernel114<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          gpu_a, *gpu_b_data, *gpu_c);
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else {
      real_T blockDims[3];
      int32_T OW;
      int32_T b_OH;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims[3];
      OH = (static_cast<uint32_T>(cpu_b->size[0]) +
            static_cast<uint32_T>(cpu_a->size[0])) -
           1U;
      nc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      mc = static_cast<int32_T>(
          std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
      blockDims[0] = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(nc) - 1.0)) /
          static_cast<real_T>(nc));
      blockDims[1] = std::floor(
          (static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                               static_cast<uint32_T>(mc)) -
           1.0) /
          static_cast<real_T>(mc));
      threadDims[0] = static_cast<int8_T>(nc);
      threadDims[1] = static_cast<int8_T>(mc);
      OW = cpu_a->size[1];
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
      if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_a->size[0];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[0] = mc - 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_expanded, k, &cg_emlrtRTEI);
      if (cpu_a->size[1] > MAX_int32_T - cpu_a->size[1]) {
        mc = MAX_int32_T;
      } else {
        mc = cpu_a->size[1] + cpu_a->size[1];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[1] = mc - 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_expanded, k, &cg_emlrtRTEI);
      k = ((static_cast<int32_T>(OH) + cpu_a->size[0]) - 1) *
              ((cpu_a->size[1] + cpu_a->size[1]) - 1) -
          1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_expanded, &b_gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel115#" MW_AT_LINE);
        conv2_kernel115<<<grid, block>>>(k, b_gpu_expanded);
      }
      nc = cpu_b->size[0];
      k = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = 1;
      b_cpu_y->size[1] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(b_cpu_y, k, &dg_emlrtRTEI);
      b_cpu_y->data[0] = 1;
      mc = 1;
      profileLoopStart("c_conv2_loop_1", __LINE__, (nc - 2) + 1, "");
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        b_cpu_y->data[k + 1] = mc;
      }
      profileLoopEnd();
      k = cpu_iv1->size[0];
      cpu_iv1->size[0] = b_cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_iv1, k, &cg_emlrtRTEI);
      k = b_cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel116#" MW_AT_LINE);
        conv2_kernel116<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv1);
      }
      mc = static_cast<int32_T>(
               std::floor(static_cast<real_T>(cpu_a->size[1]) / 2.0)) +
           static_cast<int32_T>(
               std::floor((static_cast<real_T>(cpu_a->size[1]) - 1.0) / 2.0));
      nc = cpu_iv1->size[0];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      if (*b_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel117#" MW_AT_LINE);
        conv2_kernel117<<<grid, block>>>(*gpu_b, mc, gpu_iv1, nc - 1,
                                         b_gpu_expanded,
                                         b_cpu_expanded->size[0U]);
      }
      nc = cpu_a->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_rows, k, &dg_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      profileLoopStart("c_conv2_loop_2", __LINE__, (nc - 2) + 1, "");
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      profileLoopEnd();
      nc = cpu_a->size[1];
      k = cpu_cols->size[0] * cpu_cols->size[1];
      cpu_cols->size[0] = 1;
      cpu_cols->size[1] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_cols, k, &dg_emlrtRTEI);
      cpu_cols->data[0] = 0;
      mc = 0;
      profileLoopStart("c_conv2_loop_3", __LINE__, (nc - 2) + 1, "");
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_cols->data[k + 1] = mc;
      }
      profileLoopEnd();
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      cpu_c->size[1] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_c, k, &cg_emlrtRTEI);
      if (blockDims[0] < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims[0]);
      } else {
        OH = MAX_uint32_T;
      }
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                              dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                              dim3(static_cast<uint32_T>(threadDims[0]),
                                   static_cast<uint32_T>(threadDims[1]), 1U),
                              &grid, &block);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_cols, &gpu_cols, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_cols, cpu_cols);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel118#" MW_AT_LINE);
        conv2_kernel118<<<grid, block>>>(
            b_gpu_expanded, gpu_cols, gpu_rows, gpu_a, b_OH - 1, OW - 1, *gpu_c,
            cpu_a->size[0U], cpu_a->size[1U], b_cpu_expanded->size[0U],
            cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else {
    if (*b_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
    }
    *b_outdatedOnCpu = false;
    nvtxMarkA("#c_conv2NonSeparable#" MW_AT_LINE);
    c_conv2NonSeparable((emxArray_creal_T *)cpu_a, &gpu_a, &a_outdatedOnGpu,
                        cpu_b, cpu_c, c_outdatedOnCpu, gpu_c, c_outdatedOnGpu);
  }
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_cols);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_expanded);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_expanded);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_a);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_cols);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_b_data), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_d), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_d1), __FILE__, __LINE__);
  nvtxRangePop();
}

//
//
void conv2(const emxArray_real_T *cpu_a, emxArray_real_T *cpu_b,
           boolean_T *b_outdatedOnCpu, emxArray_real_T *gpu_b,
           boolean_T *b_outdatedOnGpu, emxArray_real_T *cpu_c,
           boolean_T *c_outdatedOnCpu, emxArray_real_T *gpu_c,
           boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_cols;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_cols;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T b_gpu_a;
  emxArray_real_T b_gpu_c;
  emxArray_real_T b_gpu_expanded;
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *b_cpu_a;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *b_cpu_expanded;
  emxArray_real_T *cpu_expanded;
  boolean_T a_outdatedOnGpu;
  nvtxRangePushA("#fcn#conv2#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_cols);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_a);
  a_outdatedOnGpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_c, 1, &eg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_a, 1, &wf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_expanded, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_expanded, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_cols, 2, &cg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    if (cpu_a->size[1] < 1) {
      int32_T k;
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, k, &yf_emlrtRTEI);
      cpu_c->data[0] = 0.0;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      real_T b;
      int32_T k;
      boolean_T validLaunchParams;
      k = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_a, k, &wf_emlrtRTEI);
      k = cpu_a->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_a, &b_gpu_a, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel43#" MW_AT_LINE);
        conv2_kernel43<<<grid, block>>>(gpu_a, k, b_gpu_a, cpu_a->size[0U]);
      }
      if (*b_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      b = cpu_b->data[0];
      k = b_cpu_c->size[0];
      b_cpu_c->size[0] = b_cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_c, k, &ag_emlrtRTEI);
      k = b_cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel44#" MW_AT_LINE);
        conv2_kernel44<<<grid, block>>>(k, b_gpu_c);
      }
      if (b_cpu_a->size[0] == 1) {
        b = cpu_b->data[0];
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_c, k, &ag_emlrtRTEI);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
        nvtxMarkA("#conv2_kernel45#" MW_AT_LINE);
        conv2_kernel45<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b, b_gpu_a,
                                                                b_gpu_c);
      } else {
        int32_T b_OH;
        int32_T mc;
        int32_T nc;
        int32_T offsetH;
        int8_T threadDims[3];
        nc = static_cast<int32_T>(std::fmin(
            32.0, (static_cast<real_T>(b_cpu_a->size[0]) + 1.0) - 1.0));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(b_cpu_a->size[0]) +
                                 static_cast<uint32_T>(nc)) -
             1.0) /
            static_cast<real_T>(nc)));
        threadDims[0] = static_cast<int8_T>(nc);
        b_OH = b_cpu_a->size[0];
        if (b_cpu_a->size[0] > 2147483646) {
          mc = MAX_int32_T;
        } else {
          mc = b_cpu_a->size[0] + 1;
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_expanded, k, &cg_emlrtRTEI);
        k = b_cpu_a->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel47#" MW_AT_LINE);
          conv2_kernel47<<<grid, block>>>(k, gpu_expanded);
        }
        nc = b_cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = b_cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_y, k, &dg_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        profileLoopStart("conv2_loop_0", __LINE__, (nc - 2) + 1, "");
        for (k = 0; k <= nc - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, k, &cg_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel48#" MW_AT_LINE);
          conv2_kernel48<<<grid, block>>>(gpu_y, k, gpu_iv);
        }
        nc = cpu_iv->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel49#" MW_AT_LINE);
          conv2_kernel49<<<grid, block>>>(b_gpu_a, gpu_iv, nc - 1,
                                          gpu_expanded);
        }
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = b_cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_c, k, &cg_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims[0]), 1U, 1U), &grid, &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel50#" MW_AT_LINE);
          conv2_kernel50<<<grid, block>>>(b, gpu_expanded, b_OH - 1, b_gpu_c);
        }
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = b_cpu_c->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, k, &bg_emlrtRTEI);
      k = b_cpu_c->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel46#" MW_AT_LINE);
        conv2_kernel46<<<grid, block>>>(b_gpu_c, k, *gpu_c, cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    int32_T nc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    if (cpu_a->size[1] == 0) {
      nc = 1;
    } else {
      nc = cpu_a->size[1];
    }
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_c, k, &xf_emlrtRTEI);
    k = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2_kernel51#" MW_AT_LINE);
      conv2_kernel51<<<grid, block>>>(k, *gpu_c);
    }
    if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0)) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = nc;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, k, &xf_emlrtRTEI);
      k = mc * nc - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel52#" MW_AT_LINE);
        conv2_kernel52<<<grid, block>>>(k, *gpu_c);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else if ((cpu_b->size[0] == 1) &&
               ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1))) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, k, &xf_emlrtRTEI);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      if (*b_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      nvtxMarkA("#conv2_kernel53#" MW_AT_LINE);
      conv2_kernel53<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_a, *gpu_b,
                                                              *gpu_c);
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else {
      real_T blockDims[3];
      int32_T OW;
      int32_T b_OH;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims[3];
      OH = (static_cast<uint32_T>(cpu_b->size[0]) +
            static_cast<uint32_T>(cpu_a->size[0])) -
           1U;
      nc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      mc = static_cast<int32_T>(
          std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
      blockDims[0] = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(nc) - 1.0)) /
          static_cast<real_T>(nc));
      blockDims[1] = std::floor(
          (static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                               static_cast<uint32_T>(mc)) -
           1.0) /
          static_cast<real_T>(mc));
      threadDims[0] = static_cast<int8_T>(nc);
      threadDims[1] = static_cast<int8_T>(mc);
      OW = cpu_a->size[1];
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
      if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_a->size[0];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[0] = mc - 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_expanded, k, &cg_emlrtRTEI);
      if (cpu_a->size[1] > MAX_int32_T - cpu_a->size[1]) {
        mc = MAX_int32_T;
      } else {
        mc = cpu_a->size[1] + cpu_a->size[1];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[1] = mc - 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_expanded, k, &cg_emlrtRTEI);
      k = ((static_cast<int32_T>(OH) + cpu_a->size[0]) - 1) *
              ((cpu_a->size[1] + cpu_a->size[1]) - 1) -
          1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_expanded, &b_gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel54#" MW_AT_LINE);
        conv2_kernel54<<<grid, block>>>(k, b_gpu_expanded);
      }
      nc = cpu_b->size[0];
      k = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = 1;
      b_cpu_y->size[1] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(b_cpu_y, k, &dg_emlrtRTEI);
      b_cpu_y->data[0] = 1;
      mc = 1;
      profileLoopStart("conv2_loop_1", __LINE__, (nc - 2) + 1, "");
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        b_cpu_y->data[k + 1] = mc;
      }
      profileLoopEnd();
      k = cpu_iv1->size[0];
      cpu_iv1->size[0] = b_cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_iv1, k, &cg_emlrtRTEI);
      k = b_cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel55#" MW_AT_LINE);
        conv2_kernel55<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv1);
      }
      mc = static_cast<int32_T>(
               std::floor(static_cast<real_T>(cpu_a->size[1]) / 2.0)) +
           static_cast<int32_T>(
               std::floor((static_cast<real_T>(cpu_a->size[1]) - 1.0) / 2.0));
      nc = cpu_iv1->size[0];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      if (*b_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel56#" MW_AT_LINE);
        conv2_kernel56<<<grid, block>>>(*gpu_b, mc, gpu_iv1, nc - 1,
                                        b_gpu_expanded,
                                        b_cpu_expanded->size[0U]);
      }
      nc = cpu_a->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_rows, k, &dg_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      profileLoopStart("conv2_loop_2", __LINE__, (nc - 2) + 1, "");
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      profileLoopEnd();
      nc = cpu_a->size[1];
      k = cpu_cols->size[0] * cpu_cols->size[1];
      cpu_cols->size[0] = 1;
      cpu_cols->size[1] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_cols, k, &dg_emlrtRTEI);
      cpu_cols->data[0] = 0;
      mc = 0;
      profileLoopStart("conv2_loop_3", __LINE__, (nc - 2) + 1, "");
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_cols->data[k + 1] = mc;
      }
      profileLoopEnd();
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      cpu_c->size[1] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_c, k, &cg_emlrtRTEI);
      if (blockDims[0] < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims[0]);
      } else {
        OH = MAX_uint32_T;
      }
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                              dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                              dim3(static_cast<uint32_T>(threadDims[0]),
                                   static_cast<uint32_T>(threadDims[1]), 1U),
                              &grid, &block);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_cols, &gpu_cols, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_cols, cpu_cols);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel57#" MW_AT_LINE);
        conv2_kernel57<<<grid, block>>>(
            b_gpu_expanded, gpu_cols, gpu_rows, gpu_a, b_OH - 1, OW - 1, *gpu_c,
            cpu_a->size[0U], cpu_a->size[1U], b_cpu_expanded->size[0U],
            cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else {
    nvtxMarkA("#conv2NonSeparable#" MW_AT_LINE);
    conv2NonSeparable((emxArray_real_T *)cpu_a, &gpu_a, &a_outdatedOnGpu, cpu_b,
                      b_outdatedOnCpu, gpu_b, b_outdatedOnGpu, cpu_c,
                      c_outdatedOnCpu, gpu_c, c_outdatedOnGpu);
  }
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_cols);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_expanded);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_expanded);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_a);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_cols);
  nvtxRangePop();
}

//
//
void d_conv2(const emxArray_creal_T *cpu_a, emxArray_real_T *cpu_b,
             boolean_T *b_outdatedOnCpu, emxArray_real_T *gpu_b,
             boolean_T *b_outdatedOnGpu, emxArray_creal_T *cpu_c,
             boolean_T *c_outdatedOnCpu, emxArray_creal_T *gpu_c,
             boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_expanded;
  emxArray_creal_T gpu_a;
  emxArray_creal_T gpu_newIm;
  emxArray_creal_T *cpu_expanded;
  emxArray_creal_T *cpu_newIm;
  emxArray_int32_T b_gpu_rows;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_rows;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *b_cpu_expanded;
  nvtxRangePushA("#fcn#d_conv2#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_a);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_expanded, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_expanded, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_rows, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_newIm, 1, &cg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    int32_T k;
    k = cpu_c->size[0];
    cpu_c->size[0] = 1;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, k, &bg_emlrtRTEI);
    if (*b_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
    }
    cpu_c->data[0].re = cpu_b->data[0] * cpu_a->data[0].re;
    *b_outdatedOnCpu = false;
    cpu_c->data[0].im = cpu_b->data[0] * cpu_a->data[0].im;
    *c_outdatedOnCpu = false;
    *c_outdatedOnGpu = true;
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    k = cpu_c->size[0];
    cpu_c->size[0] = mc;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, k, &xf_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2_kernel131#" MW_AT_LINE);
      conv2_kernel131<<<grid, block>>>(mc - 1, *gpu_c);
    }
    *c_outdatedOnGpu = false;
    *c_outdatedOnCpu = true;
    if (cpu_a->size[0] != 0) {
      if ((cpu_b->size[0] == 1) && (cpu_a->size[0] == 1)) {
        creal_T a;
        real_T cv_im;
        real_T cv_re;
        a = cpu_a->data[0];
        k = cpu_c->size[0];
        cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_c, k, &xf_emlrtRTEI);
        if (*b_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
        }
        cv_im = cpu_b->data[0] * a.re;
        *b_outdatedOnCpu = false;
        cv_re = cpu_b->data[0] * a.im;
        cpu_c->data[0].im = cv_re;
        cpu_c->data[0].re = cv_im;
        *c_outdatedOnCpu = false;
        *c_outdatedOnGpu = true;
      } else {
        real_T cv_re;
        int32_T b_OH;
        int32_T n;
        int32_T offsetH;
        uint32_T OH;
        int8_T threadDims_idx_0;
        OH = (static_cast<uint32_T>(cpu_b->size[0]) +
              static_cast<uint32_T>(cpu_a->size[0])) -
             1U;
        mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
        cv_re = std::floor(
            (static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
            static_cast<real_T>(mc));
        threadDims_idx_0 = static_cast<int8_T>(mc);
        if (OH > 2147483647U) {
          OH = 2147483647U;
        }
        b_OH = static_cast<int32_T>(OH);
        offsetH = static_cast<int32_T>(
                      std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                  static_cast<int32_T>(std::floor(
                      (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
        if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
          mc = MAX_int32_T;
        } else {
          mc = static_cast<int32_T>(OH) + cpu_a->size[0];
        }
        k = b_cpu_expanded->size[0];
        b_cpu_expanded->size[0] = mc - 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_expanded, k, &cg_emlrtRTEI);
        k = (static_cast<int32_T>(OH) + cpu_a->size[0]) - 2;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel132#" MW_AT_LINE);
          conv2_kernel132<<<grid, block>>>(k, gpu_expanded);
        }
        n = cpu_b->size[0];
        k = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_b->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(b_cpu_y, k, &dg_emlrtRTEI);
        b_cpu_y->data[0] = 1;
        mc = 1;
        profileLoopStart("d_conv2_loop_1", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          b_cpu_y->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_iv1->size[0];
        cpu_iv1->size[0] = b_cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv1, k, &cg_emlrtRTEI);
        k = b_cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(b_cpu_y, &gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, b_cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel133#" MW_AT_LINE);
          conv2_kernel133<<<grid, block>>>(offsetH, gpu_y, k, gpu_iv1);
        }
        mc = cpu_iv1->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
        if (*b_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
        }
        *b_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel134#" MW_AT_LINE);
          conv2_kernel134<<<grid, block>>>(*gpu_b, gpu_iv1, mc - 1,
                                           gpu_expanded);
        }
        n = cpu_a->size[0];
        k = b_cpu_rows->size[0] * b_cpu_rows->size[1];
        b_cpu_rows->size[0] = 1;
        b_cpu_rows->size[1] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(b_cpu_rows, k, &dg_emlrtRTEI);
        b_cpu_rows->data[0] = 0;
        mc = 0;
        profileLoopStart("d_conv2_loop_3", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          b_cpu_rows->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_c->size[0];
        cpu_c->size[0] = static_cast<int32_T>(OH);
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_c, k, &cg_emlrtRTEI);
        if (cv_re < 4.294967296E+9) {
          OH = static_cast<uint32_T>(cv_re);
        } else {
          OH = MAX_uint32_T;
        }
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1), dim3(OH, 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid,
            &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(b_cpu_rows, &gpu_rows, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_a, &gpu_a, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, b_cpu_rows);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_a, cpu_a);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel135#" MW_AT_LINE);
          conv2_kernel135<<<grid, block>>>(gpu_expanded, gpu_rows, gpu_a,
                                           b_OH - 1, *gpu_c, cpu_a->size[0U]);
        }
      }
    }
  } else {
    int32_T k;
    int32_T mc;
    boolean_T validLaunchParams;
    if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
      mc = cpu_a->size[0] + cpu_b->size[0];
    } else {
      mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
    }
    k = cpu_c->size[0];
    cpu_c->size[0] = mc;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, k, &jg_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2_kernel136#" MW_AT_LINE);
      conv2_kernel136<<<grid, block>>>(mc - 1, *gpu_c);
    }
    *c_outdatedOnGpu = false;
    *c_outdatedOnCpu = true;
    if ((cpu_a->size[0] != 0) && (cpu_b->size[0] != 0)) {
      if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
        real_T cv_im;
        real_T cv_re;
        if (*b_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
        }
        *b_outdatedOnCpu = false;
        cv_re = cpu_b->data[0];
        k = cpu_c->size[0];
        cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_c, k, &jg_emlrtRTEI);
        cv_im = cv_re * cpu_a->data[0].re;
        cv_re *= cpu_a->data[0].im;
        cpu_c->data[0].im = cv_re;
        cpu_c->data[0].re = cv_im;
        *c_outdatedOnCpu = false;
        *c_outdatedOnGpu = true;
      } else {
        int32_T b_OH;
        int32_T n;
        int32_T offsetH;
        uint32_T OH;
        boolean_T c_needsGpuEnsureCapacity;
        boolean_T rows_needsGpuEnsureCapacity;
        boolean_T rows_outdatedOnGpu;
        OH = (static_cast<uint32_T>(cpu_a->size[0]) +
              static_cast<uint32_T>(cpu_b->size[0])) -
             1U;
        if (OH > 2147483647U) {
          OH = 2147483647U;
        }
        b_OH = static_cast<int32_T>(OH);
        offsetH = static_cast<int32_T>(
                      std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                  static_cast<int32_T>(std::floor(
                      (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
        if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
          mc = MAX_int32_T;
        } else {
          mc = static_cast<int32_T>(OH) + cpu_b->size[0];
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_expanded, k, &cg_emlrtRTEI);
        k = (static_cast<int32_T>(OH) + cpu_b->size[0]) - 2;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_expanded, &b_gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel137#" MW_AT_LINE);
          conv2_kernel137<<<grid, block>>>(k, b_gpu_expanded);
        }
        n = cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_y, k, &dg_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        profileLoopStart("d_conv2_loop_0", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, k, &cg_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &b_gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel138#" MW_AT_LINE);
          conv2_kernel138<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv);
        }
        mc = cpu_iv->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_a, &gpu_a, false);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_a, cpu_a);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel139#" MW_AT_LINE);
          conv2_kernel139<<<grid, block>>>(gpu_a, gpu_iv, mc - 1,
                                           b_gpu_expanded);
        }
        n = cpu_b->size[0];
        k = cpu_rows->size[0] * cpu_rows->size[1];
        cpu_rows->size[0] = 1;
        cpu_rows->size[1] = cpu_b->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_rows, k, &dg_emlrtRTEI);
        cpu_rows->data[0] = 0;
        rows_outdatedOnGpu = true;
        rows_needsGpuEnsureCapacity = true;
        mc = 0;
        profileLoopStart("d_conv2_loop_2", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          cpu_rows->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_c->size[0];
        cpu_c->size[0] = static_cast<int32_T>(OH);
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_c, k, &cg_emlrtRTEI);
        c_needsGpuEnsureCapacity = true;
        mc = cpu_b->size[0];
        profileLoopStart("d_conv2_loop_4", __LINE__, (b_OH - 1) + 1, "");
        for (offsetH = 0; offsetH < b_OH; offsetH++) {
          real_T cv_im;
          real_T cv_re;
          boolean_T newIm_outdatedOnCpu;
          k = cpu_newIm->size[0];
          cpu_newIm->size[0] = cpu_rows->size[1];
          nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
          emxEnsureCapacity_creal_T(cpu_newIm, k, &cg_emlrtRTEI);
          k = cpu_rows->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(k), &grid, &block,
                                  2147483647U);
          if (rows_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_int32_T(cpu_rows, &b_gpu_rows,
                                         !rows_outdatedOnGpu);
          }
          rows_needsGpuEnsureCapacity = false;
          nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal_T(cpu_newIm, &gpu_newIm, true);
          if (rows_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_rows, cpu_rows);
          }
          rows_outdatedOnGpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#conv2_kernel140#" MW_AT_LINE);
            conv2_kernel140<<<grid, block>>>(b_gpu_expanded, offsetH,
                                             b_gpu_rows, k, gpu_newIm);
          }
          newIm_outdatedOnCpu = true;
          cv_re = 0.0;
          cv_im = 0.0;
          profileLoopStart("d_conv2_loop_5", __LINE__, (mc - 1) + 1, "");
          for (n = 0; n < mc; n++) {
            if (newIm_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_creal_T(cpu_newIm, &gpu_newIm);
            }
            if (*b_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
            }
            cv_re +=
                cpu_newIm->data[n].re * cpu_b->data[(cpu_b->size[0] - n) - 1];
            newIm_outdatedOnCpu = false;
            *b_outdatedOnCpu = false;
            cv_im +=
                cpu_newIm->data[n].im * cpu_b->data[(cpu_b->size[0] - n) - 1];
          }
          profileLoopEnd();
          if (c_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_creal_T(cpu_c, gpu_c, true);
          }
          c_needsGpuEnsureCapacity = false;
          nvtxMarkA("#conv2_kernel141#" MW_AT_LINE);
          conv2_kernel141<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              cv_im, cv_re, offsetH, *gpu_c);
        }
        profileLoopEnd();
      }
    }
  }
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_newIm);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_expanded);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_rows);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_newIm);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (conv2.cu)

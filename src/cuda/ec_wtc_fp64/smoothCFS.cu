#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// smoothCFS.cu
//
// Code generation for function 'smoothCFS'
//

// Include files
#include "smoothCFS.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "ifft.h"
#include "mtimes.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hipfft/hipfft.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo lg_emlrtRTEI{
    15,          // lineNo
    5,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo mg_emlrtRTEI{
    16,          // lineNo
    5,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo ng_emlrtRTEI{
    18,          // lineNo
    5,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo og_emlrtRTEI{
    25,          // lineNo
    1,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo pg_emlrtRTEI{
    30,          // lineNo
    27,          // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo qg_emlrtRTEI{
    47,          // lineNo
    28,          // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo rg_emlrtRTEI{
    47,          // lineNo
    9,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo sg_emlrtRTEI{
    58,          // lineNo
    1,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo tg_emlrtRTEI{
    53,          // lineNo
    7,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo ug_emlrtRTEI{
    30,          // lineNo
    5,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo vg_emlrtRTEI{
    45,          // lineNo
    9,           // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo wg_emlrtRTEI{
    47,          // lineNo
    23,          // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

static emlrtRTEInfo xg_emlrtRTEI{
    50,          // lineNo
    23,          // colNo
    "smoothCFS", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "smoothCFS.m" // pName
};

// Function Declarations
static void binary_expand_op_8(emxArray_creal_T *in1,
                               const emxArray_real_T *in2,
                               const emxArray_creal_T *in3);

static __global__ void smoothCFS_kernel142(const real_T d,
                                           const emxArray_int32_T omega_tmp1,
                                           const int32_T b_omega_tmp1,
                                           emxArray_real_T omega_tmp2);

static __global__ void smoothCFS_kernel143(emxArray_real_T omega);

static __global__ void smoothCFS_kernel144(const emxArray_real_T omega_tmp2,
                                           const int32_T b,
                                           emxArray_real_T omega);

static __global__ void
smoothCFS_kernel145(const int32_T offsetH, const int32_T midpoint,
                    const emxArray_real_T omega_tmp2, const int32_T N,
                    emxArray_real_T omega, int32_T omega_tmp2_dim1);

static __global__ void smoothCFS_kernel146(const int32_T cfs,
                                           emxArray_creal_T cfsDFT);

static __global__ void
smoothCFS_kernel147(const emxArray_real_T scales, const emxArray_real_T omega,
                    const int32_T b_scales, const int32_T b,
                    emxArray_real_T Fmat, int32_T Fmat_dim0);

static __global__ void smoothCFS_kernel148(const emxArray_creal_T cfsDFT,
                                           const emxArray_real_T Fmat,
                                           const int32_T b_Fmat,
                                           emxArray_creal_T c_Fmat);

static __global__ void smoothCFS_kernel149(const emxArray_creal_T r,
                                           const int32_T b_r,
                                           emxArray_real_T smooth);

static __global__ void smoothCFS_kernel150(const emxArray_real_T smooth,
                                           const int32_T b, emxArray_real_T a,
                                           int32_T smooth_dim0);

static __global__ void smoothCFS_kernel151(const int32_T a, emxArray_real_T c);

static __global__ void smoothCFS_kernel152(const emxArray_real_T c,
                                           const int32_T b_c,
                                           emxArray_real_T cfs,
                                           int32_T cfs_dim0);

static __global__ void smoothCFS_kernel153(const real_T npad,
                                           const emxArray_real_T a,
                                           emxArray_real_T c);

static __global__ void smoothCFS_kernel154(const int32_T a,
                                           emxArray_real_T expanded);

static __global__ void smoothCFS_kernel155(const emxArray_int32_T y,
                                           const int32_T b_y,
                                           emxArray_int32_T iv1);

static __global__ void smoothCFS_kernel156(const emxArray_real_T a,
                                           const emxArray_int32_T iv1,
                                           const int32_T b,
                                           emxArray_real_T expanded);

static __global__ void smoothCFS_kernel157(const real_T npad,
                                           const emxArray_real_T expanded,
                                           const int32_T b, emxArray_real_T c);

static __global__ void smoothCFS_kernel158(const int32_T smooth,
                                           const int32_T b_smooth,
                                           emxArray_real_T cfs,
                                           int32_T cfs_dim0);

static __global__ void smoothCFS_kernel159(const emxArray_real_T smooth,
                                           emxArray_real_T b_smooth);

static __global__ void smoothCFS_kernel160(const real_T npad, const int32_T b,
                                           real_T a_data[1]);

static __global__ void smoothCFS_kernel161(const int32_T smooth,
                                           emxArray_real_T expanded);

static __global__ void smoothCFS_kernel162(const int32_T offsetH,
                                           const emxArray_int32_T y,
                                           const int32_T b_y,
                                           emxArray_int32_T iv);

static __global__ void smoothCFS_kernel163(const emxArray_int32_T y,
                                           const int32_T b_y,
                                           emxArray_int32_T iv2);

static __global__ void
smoothCFS_kernel164(const emxArray_real_T smooth, const int32_T b_smooth,
                    const int32_T b, emxArray_real_T c_smooth,
                    int32_T smooth_dim0, int32_T b_smooth_dim0);

static __global__ void
smoothCFS_kernel165(const emxArray_real_T smooth, const int32_T smooth_dim0,
                    const emxArray_int32_T iv2, const emxArray_int32_T iv,
                    const int32_T b_smooth, const int32_T c_smooth,
                    emxArray_real_T expanded, int32_T expanded_dim0);

static __global__ void
smoothCFS_kernel166(const real_T npad, const emxArray_real_T expanded,
                    const emxArray_int32_T rows, const real_T ns,
                    const int32_T b, const int32_T c, emxArray_real_T cfs,
                    int32_T expanded_dim0, int32_T cfs_dim0);

static __global__ void smoothCFS_kernel167(const int32_T cfs,
                                           emxArray_creal_T cfsDFT);

static __global__ void smoothCFS_kernel168(const emxArray_real_T cfs,
                                           const int32_T b, const int32_T c,
                                           emxArray_creal_T cfsDFT,
                                           int32_T cfsDFT_dim0,
                                           int32_T cfs_dim0);

static __global__ void smoothCFS_kernel169(const emxArray_real_T cfs,
                                           const int32_T b_cfs,
                                           emxArray_creal_T cfsDFT);

static __global__ void smoothCFS_kernel170(const int32_T N,
                                           const int32_T midpoint,
                                           emxArray_creal_T cfsDFT);

static __global__ void smoothCFS_kernel171(const int32_T N,
                                           const int32_T cfsDFT,
                                           const int32_T midpoint,
                                           emxArray_creal_T b_cfsDFT,
                                           int32_T cfsDFT_dim0);

static __global__ void smoothCFS_kernel172(const int32_T b,
                                           emxArray_int32_T omega_tmp1);

static __global__ void smoothCFS_kernel173(const real_T cv_im,
                                           const emxArray_int32_T omega_tmp1,
                                           const int32_T b_omega_tmp1,
                                           emxArray_real_T omega_tmp2);

static __global__ void smoothCFS_kernel174(emxArray_real_T omega);

static __global__ void smoothCFS_kernel175(const emxArray_real_T omega_tmp2,
                                           const int32_T b,
                                           emxArray_real_T omega);

static __global__ void
smoothCFS_kernel176(const int32_T offsetH, const int32_T x,
                    const emxArray_real_T omega_tmp2, const int32_T ySize_idx_1,
                    emxArray_real_T omega, int32_T omega_tmp2_dim1);

static __global__ void smoothCFS_kernel177(const int32_T cfs,
                                           emxArray_creal_T cfsDFT);

static __global__ void
smoothCFS_kernel178(const emxArray_real_T scales, const emxArray_real_T omega,
                    const int32_T b_scales, const int32_T b,
                    emxArray_real_T Fmat, int32_T Fmat_dim0);

static __global__ void smoothCFS_kernel179(const emxArray_creal_T cfsDFT,
                                           const emxArray_real_T Fmat,
                                           const int32_T b_Fmat,
                                           emxArray_creal_T c_Fmat);

static __global__ void smoothCFS_kernel180(const emxArray_creal_T smooth,
                                           const int32_T b, emxArray_creal_T a,
                                           int32_T smooth_dim0);

static __global__ void smoothCFS_kernel181(const int32_T a, emxArray_creal_T c);

static __global__ void smoothCFS_kernel182(const emxArray_creal_T c,
                                           const int32_T b_c,
                                           emxArray_creal_T cfs,
                                           int32_T cfs_dim0);

static __global__ void smoothCFS_kernel183(const emxArray_creal_T a,
                                           const real_T b_a, real_T *d,
                                           real_T *d1);

static __global__ void smoothCFS_kernel184(const real_T *d1, const real_T *d,
                                           emxArray_creal_T c);

static __global__ void smoothCFS_kernel185(const int32_T a,
                                           emxArray_creal_T expanded);

static __global__ void smoothCFS_kernel186(const emxArray_int32_T y,
                                           const int32_T b_y,
                                           emxArray_int32_T iv1);

static __global__ void smoothCFS_kernel187(const emxArray_creal_T a,
                                           const emxArray_int32_T iv1,
                                           const int32_T b,
                                           emxArray_creal_T expanded);

static __global__ void smoothCFS_kernel188(const emxArray_creal_T expanded,
                                           const real_T a, const int32_T b,
                                           emxArray_creal_T c);

static __global__ void smoothCFS_kernel189(const int32_T smooth,
                                           const int32_T b_smooth,
                                           emxArray_creal_T cfs,
                                           int32_T cfs_dim0);

static __global__ void smoothCFS_kernel190(const real_T a, const int32_T b,
                                           creal_T a_data[1]);

static __global__ void smoothCFS_kernel191(const int32_T smooth,
                                           emxArray_creal_T expanded);

static __global__ void smoothCFS_kernel192(const int32_T offsetH,
                                           const emxArray_int32_T y,
                                           const int32_T b_y,
                                           emxArray_int32_T iv);

static __global__ void smoothCFS_kernel193(const emxArray_int32_T y,
                                           const int32_T b_y,
                                           emxArray_int32_T iv2);

static __global__ void
smoothCFS_kernel194(const emxArray_creal_T smooth, const int32_T b_smooth,
                    const int32_T b, emxArray_creal_T c_smooth,
                    int32_T smooth_dim0, int32_T b_smooth_dim0);

static __global__ void
smoothCFS_kernel195(const emxArray_creal_T smooth, const int32_T smooth_dim0,
                    const emxArray_int32_T iv2, const emxArray_int32_T iv,
                    const int32_T b_smooth, const int32_T c_smooth,
                    emxArray_creal_T expanded, int32_T expanded_dim0);

static __global__ void
smoothCFS_kernel196(const int32_T offsetH, const emxArray_creal_T expanded,
                    const int32_T k, const emxArray_int32_T rows,
                    const int32_T b_rows, emxArray_creal_T newIm,
                    int32_T expanded_dim0);

static __global__ void
smoothCFS_kernel197(const real_T npad, const int32_T offsetH, const int32_T k,
                    emxArray_creal_T cfs, int32_T cfs_dim0);

static __global__ void
smoothCFS_kernel198(const real_T cv_im, const int32_T offsetH, const int32_T k,
                    emxArray_creal_T cfs, int32_T cfs_dim0);

static __global__ void smoothCFS_kernel199(const int32_T cfs,
                                           emxArray_creal_T cfsDFT);

static __global__ void smoothCFS_kernel200(const emxArray_creal_T cfs,
                                           const int32_T b, const int32_T c,
                                           emxArray_creal_T cfsDFT,
                                           int32_T cfsDFT_dim0,
                                           int32_T cfs_dim0);

static __global__ void smoothCFS_kernel201(const emxArray_creal_T cfs,
                                           const int32_T b_cfs,
                                           emxArray_creal_T cfsDFT);

static __global__ void smoothCFS_kernel202(const int32_T b,
                                           emxArray_int32_T omega_tmp1);

// Function Definitions
static void binary_expand_op_8(emxArray_creal_T *in1,
                               const emxArray_real_T *in2,
                               const emxArray_creal_T *in3)
{
  emxArray_creal_T *b_in2;
  int32_T aux_0_1;
  int32_T aux_1_1;
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_0_1;
  int32_T stride_1_0;
  int32_T stride_1_1;
  nvtxRangePushA("#fcn#binary_expand_op_8#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_in2, 2, &qg_emlrtRTEI, true);
  i = b_in2->size[0] * b_in2->size[1];
  if (in3->size[0] == 1) {
    b_in2->size[0] = in2->size[0];
  } else {
    b_in2->size[0] = in3->size[0];
  }
  if (in3->size[1] == 1) {
    b_in2->size[1] = in2->size[1];
  } else {
    b_in2->size[1] = in3->size[1];
  }
  nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
  emxEnsureCapacity_creal_T(b_in2, i, &qg_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_0 = (in3->size[0] != 1);
  stride_1_1 = (in3->size[1] != 1);
  aux_0_1 = 0;
  aux_1_1 = 0;
  if (in3->size[1] == 1) {
    b = in2->size[1];
  } else {
    b = in3->size[1];
  }
  profileLoopStart("binary_expand_op_8_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    int32_T c;
    if (in3->size[0] == 1) {
      c = in2->size[0];
    } else {
      c = in3->size[0];
    }
    profileLoopStart("binary_expand_op_8_loop_1", __LINE__, (c - 1) + 1, "");
    for (int32_T i1{0}; i1 < c; i1++) {
      b_in2->data[i1 + b_in2->size[0] * i].re =
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1] *
          in3->data[i1 * stride_1_0 + in3->size[0] * aux_1_1].re;
      b_in2->data[i1 + b_in2->size[0] * i].im =
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1] *
          in3->data[i1 * stride_1_0 + in3->size[0] * aux_1_1].im;
    }
    profileLoopEnd();
    aux_1_1 += stride_1_1;
    aux_0_1 += stride_0_1;
  }
  profileLoopEnd();
  nvtxMarkA("#ifft#" MW_AT_LINE);
  coder::ifft(b_in2, in1);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_in2);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel142(
    const real_T d, const emxArray_int32_T omega_tmp1,
    const int32_T b_omega_tmp1, emxArray_real_T omega_tmp2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_omega_tmp1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    omega_tmp2.data[j] = static_cast<real_T>(omega_tmp1.data[j]) * d;
  }
}

static __global__
    __launch_bounds__(32, 1) void smoothCFS_kernel143(emxArray_real_T omega)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    omega.data[0] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel144(
    const emxArray_real_T omega_tmp2, const int32_T b, emxArray_real_T omega)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    omega.data[j + 1] = omega_tmp2.data[j];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel145(
    const int32_T offsetH, const int32_T midpoint,
    const emxArray_real_T omega_tmp2, const int32_T N, emxArray_real_T omega,
    int32_T omega_tmp2_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(N);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    omega.data[(j + omega_tmp2_dim1) + 1] =
        -omega_tmp2.data[midpoint + offsetH * j];
  }
}

static __global__
    __launch_bounds__(1024, 1) void smoothCFS_kernel146(const int32_T cfs,
                                                        emxArray_creal_T cfsDFT)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(cfs);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    cfsDFT.data[j].re = 0.0;
    cfsDFT.data[j].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel147(
    const emxArray_real_T scales, const emxArray_real_T omega,
    const int32_T b_scales, const int32_T b, emxArray_real_T Fmat,
    int32_T Fmat_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b) + 1UL) *
                (static_cast<uint64_T>(b_scales) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_scales) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_scales) + 1UL));
    b_cv = omega.data[j] * scales.data[i];
    Fmat.data[i + Fmat_dim0 * j] = exp(-0.5 * (b_cv * b_cv));
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel148(
    const emxArray_creal_T cfsDFT, const emxArray_real_T Fmat,
    const int32_T b_Fmat, emxArray_creal_T c_Fmat)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_Fmat);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    c_Fmat.data[j].re = Fmat.data[j] * cfsDFT.data[j].re;
    c_Fmat.data[j].im = Fmat.data[j] * cfsDFT.data[j].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel149(
    const emxArray_creal_T r, const int32_T b_r, emxArray_real_T smooth)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    smooth.data[j] = r.data[j].re;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel150(
    const emxArray_real_T smooth, const int32_T b, emxArray_real_T a,
    int32_T smooth_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    a.data[j] = smooth.data[smooth_dim0 * j];
  }
}

static __global__
    __launch_bounds__(1024, 1) void smoothCFS_kernel151(const int32_T a,
                                                        emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    c.data[j] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel152(
    const emxArray_real_T c, const int32_T b_c, emxArray_real_T cfs,
    int32_T cfs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    cfs.data[cfs_dim0 * j] = c.data[j];
  }
}

static __global__ __launch_bounds__(32, 1) void smoothCFS_kernel153(
    const real_T npad, const emxArray_real_T a, emxArray_real_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = a.data[0] * npad;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel154(
    const int32_T a, emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    expanded.data[j] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel155(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    iv1.data[j] = y.data[j] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel156(
    const emxArray_real_T a, const emxArray_int32_T iv1, const int32_T b,
    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    expanded.data[iv1.data[j]] = a.data[j];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel157(
    const real_T npad, const emxArray_real_T expanded, const int32_T b,
    emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow] = expanded.data[orow] * npad;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel158(
    const int32_T smooth, const int32_T b_smooth, emxArray_real_T cfs,
    int32_T cfs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_smooth) + 1UL) *
                (static_cast<uint64_T>(smooth) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(smooth) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(smooth) + 1UL));
    cfs.data[i + cfs_dim0 * j] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void smoothCFS_kernel159(
    const emxArray_real_T smooth, emxArray_real_T b_smooth)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_smooth.data[0] = smooth.data[0];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel160(
    const real_T npad, const int32_T b, real_T a_data[1])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    a_data[j] = npad;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel161(
    const int32_T smooth, emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(smooth);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    expanded.data[j] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel162(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    int32_T q1;
    j = static_cast<int32_T>(idx);
    q1 = y.data[j];
    if ((offsetH < 0) && (q1 < MIN_int32_T - offsetH)) {
      q1 = MIN_int32_T;
    } else if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[j] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel163(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    iv2.data[j] = y.data[j] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel164(
    const emxArray_real_T smooth, const int32_T b_smooth, const int32_T b,
    emxArray_real_T c_smooth, int32_T smooth_dim0, int32_T b_smooth_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b) + 1UL) *
                (static_cast<uint64_T>(b_smooth) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_smooth) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_smooth) + 1UL));
    c_smooth.data[i + smooth_dim0 * j] = smooth.data[i + b_smooth_dim0 * j];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel165(
    const emxArray_real_T smooth, const int32_T smooth_dim0,
    const emxArray_int32_T iv2, const emxArray_int32_T iv,
    const int32_T b_smooth, const int32_T c_smooth, emxArray_real_T expanded,
    int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_smooth) + 1UL) *
                (static_cast<uint64_T>(b_smooth) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_smooth) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_smooth) + 1UL));
    expanded.data[iv.data[i] + expanded_dim0 * iv2.data[j]] =
        smooth.data[i + smooth_dim0 * j];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel166(
    const real_T npad, const emxArray_real_T expanded,
    const emxArray_int32_T rows, const real_T ns, const int32_T b,
    const int32_T c, emxArray_real_T cfs, int32_T expanded_dim0,
    int32_T cfs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T i;
    int32_T j;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                             (static_cast<uint64_T>(b) + 1UL));
    b_cv = 0.0;
    j = static_cast<int32_T>(ns);
    for (int32_T m{0}; m < j; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[(q1 + expanded_dim0 * i) - 1] * npad;
    }
    cfs.data[orow + cfs_dim0 * i] = b_cv;
  }
}

static __global__
    __launch_bounds__(1024, 1) void smoothCFS_kernel167(const int32_T cfs,
                                                        emxArray_creal_T cfsDFT)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(cfs);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    cfsDFT.data[j].re = 0.0;
    cfsDFT.data[j].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel168(
    const emxArray_real_T cfs, const int32_T b, const int32_T c,
    emxArray_creal_T cfsDFT, int32_T cfsDFT_dim0, int32_T cfs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    j = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(j)) /
                             (static_cast<uint64_T>(b) + 1UL));
    cfsDFT.data[j + cfsDFT_dim0 * i].re = cfs.data[j + cfs_dim0 * i];
    cfsDFT.data[j + cfsDFT_dim0 * i].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel169(
    const emxArray_real_T cfs, const int32_T b_cfs, emxArray_creal_T cfsDFT)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_cfs);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    cfsDFT.data[j].re = cfs.data[j];
    cfsDFT.data[j].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel170(
    const int32_T N, const int32_T midpoint, emxArray_creal_T cfsDFT)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(midpoint);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    cfsDFT.data[(N - i) - 1].re = cfsDFT.data[i + 1].re;
    cfsDFT.data[(N - i) - 1].im = -cfsDFT.data[i + 1].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel171(
    const int32_T N, const int32_T cfsDFT, const int32_T midpoint,
    emxArray_creal_T b_cfsDFT, int32_T cfsDFT_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(midpoint) + 1UL) *
                (static_cast<uint64_T>(cfsDFT) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    j = static_cast<int32_T>(idx % (static_cast<uint64_T>(cfsDFT) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(j)) /
                             (static_cast<uint64_T>(cfsDFT) + 1UL));
    b_cfsDFT.data[j + cfsDFT_dim0 * ((N - i) - 1)].re =
        b_cfsDFT.data[j + cfsDFT_dim0 * (i + 1)].re;
    b_cfsDFT.data[j + cfsDFT_dim0 * ((N - i) - 1)].im =
        -b_cfsDFT.data[j + cfsDFT_dim0 * (i + 1)].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel172(
    const int32_T b, emxArray_int32_T omega_tmp1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    j = static_cast<int32_T>(idx);
    omega_tmp1.data[j] = j + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel173(
    const real_T cv_im, const emxArray_int32_T omega_tmp1,
    const int32_T b_omega_tmp1, emxArray_real_T omega_tmp2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_omega_tmp1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    omega_tmp2.data[b] = static_cast<real_T>(omega_tmp1.data[b]) * cv_im;
  }
}

static __global__
    __launch_bounds__(32, 1) void smoothCFS_kernel174(emxArray_real_T omega)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    omega.data[0] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel175(
    const emxArray_real_T omega_tmp2, const int32_T b, emxArray_real_T omega)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_b;
    b_b = static_cast<int32_T>(idx);
    omega.data[b_b + 1] = omega_tmp2.data[b_b];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel176(
    const int32_T offsetH, const int32_T x, const emxArray_real_T omega_tmp2,
    const int32_T ySize_idx_1, emxArray_real_T omega, int32_T omega_tmp2_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(ySize_idx_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    omega.data[(b + omega_tmp2_dim1) + 1] = -omega_tmp2.data[x + offsetH * b];
  }
}

static __global__
    __launch_bounds__(1024, 1) void smoothCFS_kernel177(const int32_T cfs,
                                                        emxArray_creal_T cfsDFT)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(cfs);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    cfsDFT.data[b].re = 0.0;
    cfsDFT.data[b].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel178(
    const emxArray_real_T scales, const emxArray_real_T omega,
    const int32_T b_scales, const int32_T b, emxArray_real_T Fmat,
    int32_T Fmat_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b) + 1UL) *
                (static_cast<uint64_T>(b_scales) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_a;
    int32_T a;
    int32_T b_b;
    b_b = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_scales) + 1UL));
    a = static_cast<int32_T>((idx - static_cast<uint64_T>(b_b)) /
                             (static_cast<uint64_T>(b_scales) + 1UL));
    b_a = omega.data[a] * scales.data[b_b];
    Fmat.data[b_b + Fmat_dim0 * a] = exp(-0.5 * (b_a * b_a));
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel179(
    const emxArray_creal_T cfsDFT, const emxArray_real_T Fmat,
    const int32_T b_Fmat, emxArray_creal_T c_Fmat)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_Fmat);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    c_Fmat.data[b].re = Fmat.data[b] * cfsDFT.data[b].re;
    c_Fmat.data[b].im = Fmat.data[b] * cfsDFT.data[b].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel180(
    const emxArray_creal_T smooth, const int32_T b, emxArray_creal_T a,
    int32_T smooth_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_b;
    b_b = static_cast<int32_T>(idx);
    a.data[b_b] = smooth.data[smooth_dim0 * b_b];
  }
}

static __global__
    __launch_bounds__(1024, 1) void smoothCFS_kernel181(const int32_T a,
                                                        emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    c.data[b].re = 0.0;
    c.data[b].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel182(
    const emxArray_creal_T c, const int32_T b_c, emxArray_creal_T cfs,
    int32_T cfs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    cfs.data[cfs_dim0 * b] = c.data[b];
  }
}

static __global__ __launch_bounds__(32, 1) void smoothCFS_kernel183(
    const emxArray_creal_T a, const real_T b_a, real_T *d, real_T *d1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *d1 = b_a * a.data[0].re;
    *d = b_a * a.data[0].im;
  }
}

static __global__ __launch_bounds__(32, 1) void smoothCFS_kernel184(
    const real_T *d1, const real_T *d, emxArray_creal_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0].im = *d;
    c.data[0].re = *d1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel185(
    const int32_T a, emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    expanded.data[b].re = 0.0;
    expanded.data[b].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel186(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    iv1.data[b] = y.data[b] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel187(
    const emxArray_creal_T a, const emxArray_int32_T iv1, const int32_T b,
    emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_b;
    b_b = static_cast<int32_T>(idx);
    expanded.data[iv1.data[b_b]] = a.data[b_b];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel188(
    const emxArray_creal_T expanded, const real_T a, const int32_T b,
    emxArray_creal_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_a;
    b_a = static_cast<int32_T>(idx);
    c.data[b_a].re = a * expanded.data[b_a].re;
    c.data[b_a].im = a * expanded.data[b_a].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel189(
    const int32_T smooth, const int32_T b_smooth, emxArray_creal_T cfs,
    int32_T cfs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_smooth) + 1UL) *
                (static_cast<uint64_T>(smooth) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T a;
    int32_T b;
    a = static_cast<int32_T>(idx % (static_cast<uint64_T>(smooth) + 1UL));
    b = static_cast<int32_T>((idx - static_cast<uint64_T>(a)) /
                             (static_cast<uint64_T>(smooth) + 1UL));
    cfs.data[a + cfs_dim0 * b].re = 0.0;
    cfs.data[a + cfs_dim0 * b].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel190(
    const real_T a, const int32_T b, creal_T a_data[1])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_b;
    b_b = static_cast<int32_T>(idx);
    a_data[b_b].re = a;
    a_data[b_b].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel191(
    const int32_T smooth, emxArray_creal_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(smooth);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    expanded.data[b].re = 0.0;
    expanded.data[b].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel192(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T a;
    int32_T b;
    b = static_cast<int32_T>(idx);
    a = y.data[b];
    if ((offsetH < 0) && (a < MIN_int32_T - offsetH)) {
      a = MIN_int32_T;
    } else if ((offsetH > 0) && (a > MAX_int32_T - offsetH)) {
      a = MAX_int32_T;
    } else {
      a += offsetH;
    }
    iv.data[b] = a - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel193(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    iv2.data[b] = y.data[b] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel194(
    const emxArray_creal_T smooth, const int32_T b_smooth, const int32_T b,
    emxArray_creal_T c_smooth, int32_T smooth_dim0, int32_T b_smooth_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b) + 1UL) *
                (static_cast<uint64_T>(b_smooth) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T a;
    int32_T b_b;
    a = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_smooth) + 1UL));
    b_b = static_cast<int32_T>((idx - static_cast<uint64_T>(a)) /
                               (static_cast<uint64_T>(b_smooth) + 1UL));
    c_smooth.data[a + smooth_dim0 * b_b] = smooth.data[a + b_smooth_dim0 * b_b];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel195(
    const emxArray_creal_T smooth, const int32_T smooth_dim0,
    const emxArray_int32_T iv2, const emxArray_int32_T iv,
    const int32_T b_smooth, const int32_T c_smooth, emxArray_creal_T expanded,
    int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_smooth) + 1UL) *
                (static_cast<uint64_T>(b_smooth) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T a;
    int32_T b;
    a = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_smooth) + 1UL));
    b = static_cast<int32_T>((idx - static_cast<uint64_T>(a)) /
                             (static_cast<uint64_T>(b_smooth) + 1UL));
    expanded.data[iv.data[a] + expanded_dim0 * iv2.data[b]] =
        smooth.data[a + smooth_dim0 * b];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel196(
    const int32_T offsetH, const emxArray_creal_T expanded, const int32_T k,
    const emxArray_int32_T rows, const int32_T b_rows, emxArray_creal_T newIm,
    int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_rows);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T a;
    int32_T b;
    b = static_cast<int32_T>(idx);
    a = rows.data[b];
    if (a > 2147483646 - k) {
      a = MAX_int32_T;
    } else {
      a = (k + a) + 1;
    }
    newIm.data[b] = expanded.data[(a + expanded_dim0 * offsetH) - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void smoothCFS_kernel197(
    const real_T npad, const int32_T offsetH, const int32_T k,
    emxArray_creal_T cfs, int32_T cfs_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    cfs.data[k + cfs_dim0 * offsetH].re = npad;
  }
}

static __global__ __launch_bounds__(32, 1) void smoothCFS_kernel198(
    const real_T cv_im, const int32_T offsetH, const int32_T k,
    emxArray_creal_T cfs, int32_T cfs_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    cfs.data[k + cfs_dim0 * offsetH].im = cv_im;
  }
}

static __global__
    __launch_bounds__(1024, 1) void smoothCFS_kernel199(const int32_T cfs,
                                                        emxArray_creal_T cfsDFT)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(cfs);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    cfsDFT.data[b].re = 0.0;
    cfsDFT.data[b].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel200(
    const emxArray_creal_T cfs, const int32_T b, const int32_T c,
    emxArray_creal_T cfsDFT, int32_T cfsDFT_dim0, int32_T cfs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T a;
    int32_T b_b;
    b_b = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    a = static_cast<int32_T>((idx - static_cast<uint64_T>(b_b)) /
                             (static_cast<uint64_T>(b) + 1UL));
    cfsDFT.data[b_b + cfsDFT_dim0 * a] = cfs.data[b_b + cfs_dim0 * a];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel201(
    const emxArray_creal_T cfs, const int32_T b_cfs, emxArray_creal_T cfsDFT)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_cfs);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b;
    b = static_cast<int32_T>(idx);
    cfsDFT.data[b] = cfs.data[b];
  }
}

static __global__ __launch_bounds__(1024, 1) void smoothCFS_kernel202(
    const int32_T b, emxArray_int32_T omega_tmp1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_b;
    b_b = static_cast<int32_T>(idx);
    omega_tmp1.data[b_b] = b_b + 1;
  }
}

//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
void b_smoothCFS(emxArray_creal_T *cpu_cfs, boolean_T *cfs_outdatedOnCpu,
                 emxArray_creal_T *gpu_cfs, boolean_T *cfs_outdatedOnGpu,
                 emxArray_real_T *cpu_scales, emxArray_real_T *gpu_scales,
                 boolean_T *scales_outdatedOnGpu, real_T ns)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_Fmat;
  emxArray_creal_T b_gpu_expanded;
  emxArray_creal_T b_gpu_smooth;
  emxArray_creal_T gpu_a;
  emxArray_creal_T gpu_c;
  emxArray_creal_T gpu_cfsDFT;
  emxArray_creal_T gpu_expanded;
  emxArray_creal_T gpu_newIm;
  emxArray_creal_T gpu_smooth;
  emxArray_creal_T *b_cpu_Fmat;
  emxArray_creal_T *b_cpu_expanded;
  emxArray_creal_T *b_cpu_smooth;
  emxArray_creal_T *cpu_a;
  emxArray_creal_T *cpu_c;
  emxArray_creal_T *cpu_cfsDFT;
  emxArray_creal_T *cpu_expanded;
  emxArray_creal_T *cpu_newIm;
  emxArray_creal_T *cpu_smooth;
  emxArray_creal_T *smooth;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T c_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_iv2;
  emxArray_int32_T gpu_omega_tmp1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *c_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_iv2;
  emxArray_int32_T *cpu_omega_tmp1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T gpu_Fmat;
  emxArray_real_T gpu_omega;
  emxArray_real_T gpu_omega_tmp2;
  emxArray_real_T *cpu_Fmat;
  emxArray_real_T *cpu_omega;
  emxArray_real_T *cpu_omega_tmp2;
  creal_T(*gpu_a_data)[1];
  real_T a;
  real_T npad;
  real_T *gpu_d;
  real_T *gpu_d1;
  int32_T i;
  int32_T i2;
  int32_T offsetH;
  int32_T x;
  int32_T ySize_idx_1;
  boolean_T c_omega_tmp1_needsGpuEnsureCapa;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#b_smoothCFS#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_d1, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_d, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a_data, 16UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_smooth);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv2);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_c);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_a);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_Fmat);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_smooth);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_Fmat);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfsDFT);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_omega);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_omega_tmp2);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_omega_tmp1);
  c_omega_tmp1_needsGpuEnsureCapa = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  npad = std::frexp(static_cast<real_T>(cpu_cfs->size[1]), &ySize_idx_1);
  if (npad == 0.5) {
    ySize_idx_1--;
  }
  nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
  npad = rt_powd_snf(2.0, static_cast<real_T>(ySize_idx_1));
  ySize_idx_1 = static_cast<int32_T>(std::trunc(npad / 2.0));
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_omega_tmp1, 2, &lg_emlrtRTEI, true);
  if (ySize_idx_1 < 1) {
    cpu_omega_tmp1->size[0] = 1;
    cpu_omega_tmp1->size[1] = 0;
  } else {
    i = cpu_omega_tmp1->size[0] * cpu_omega_tmp1->size[1];
    cpu_omega_tmp1->size[0] = 1;
    cpu_omega_tmp1->size[1] = ySize_idx_1;
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_omega_tmp1, i, &lg_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(ySize_idx_1 - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_omega_tmp1, &gpu_omega_tmp1, true);
    c_omega_tmp1_needsGpuEnsureCapa = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel202#" MW_AT_LINE);
      smoothCFS_kernel202<<<grid, block>>>(ySize_idx_1 - 1, gpu_omega_tmp1);
    }
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_omega_tmp2, 2, &mg_emlrtRTEI, true);
  i = cpu_omega_tmp2->size[0] * cpu_omega_tmp2->size[1];
  cpu_omega_tmp2->size[0] = 1;
  cpu_omega_tmp2->size[1] = cpu_omega_tmp1->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_omega_tmp2, i, &mg_emlrtRTEI);
  i = cpu_omega_tmp1->size[1] - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  if (c_omega_tmp1_needsGpuEnsureCapa) {
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_omega_tmp1, &gpu_omega_tmp1, true);
  }
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_omega_tmp2, &gpu_omega_tmp2, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel173#" MW_AT_LINE);
    smoothCFS_kernel173<<<grid, block>>>(6.2831853071795862 / npad,
                                         gpu_omega_tmp1, i, gpu_omega_tmp2);
  }
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_omega_tmp1);
  ySize_idx_1 = static_cast<int32_T>(std::trunc((npad - 1.0) / 2.0));
  if (ySize_idx_1 < 1) {
    x = 0;
    offsetH = 1;
    i2 = -1;
  } else {
    x = ySize_idx_1 - 1;
    offsetH = -1;
    i2 = 0;
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_omega, 2, &ng_emlrtRTEI, true);
  i = cpu_omega->size[0] * cpu_omega->size[1];
  cpu_omega->size[0] = 1;
  nvtxMarkA("#div_s32#" MW_AT_LINE);
  cpu_omega->size[1] = (cpu_omega_tmp2->size[1] + div_s32(i2 - x, offsetH)) + 2;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_omega, i, &ng_emlrtRTEI);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_omega, &gpu_omega, true);
  nvtxMarkA("#smoothCFS_kernel174#" MW_AT_LINE);
  smoothCFS_kernel174<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_omega);
  ySize_idx_1 = cpu_omega_tmp2->size[1];
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(ySize_idx_1 - 1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel175#" MW_AT_LINE);
    smoothCFS_kernel175<<<grid, block>>>(gpu_omega_tmp2, ySize_idx_1 - 1,
                                         gpu_omega);
  }
  nvtxMarkA("#div_s32#" MW_AT_LINE);
  ySize_idx_1 = div_s32(i2 - x, offsetH);
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(ySize_idx_1), &grid, &block,
                          2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel176#" MW_AT_LINE);
    smoothCFS_kernel176<<<grid, block>>>(offsetH, x, gpu_omega_tmp2,
                                         ySize_idx_1, gpu_omega,
                                         cpu_omega_tmp2->size[1U]);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_omega_tmp2);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfsDFT, 2, &og_emlrtRTEI, true);
  if ((cpu_cfs->size[0] == 0) || (cpu_cfs->size[1] == 0) ||
      (static_cast<int32_T>(npad) == 0)) {
    i = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
    cpu_cfsDFT->size[0] = cpu_cfs->size[0];
    cpu_cfsDFT->size[1] = static_cast<int32_T>(npad);
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_cfsDFT, i, &og_emlrtRTEI);
    i = cpu_cfs->size[0] * static_cast<int32_T>(npad) - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel177#" MW_AT_LINE);
      smoothCFS_kernel177<<<grid, block>>>(i, gpu_cfsDFT);
    }
  } else if (cpu_cfs->size[1] != static_cast<int32_T>(npad)) {
    hipfftHandle b_fftPlanHandle;
    if (cpu_cfs->size[1] < static_cast<int32_T>(npad)) {
      i = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
      cpu_cfsDFT->size[0] = cpu_cfs->size[0];
      cpu_cfsDFT->size[1] = static_cast<int32_T>(npad);
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsDFT, i, &og_emlrtRTEI);
      i = cpu_cfs->size[0] * static_cast<int32_T>(npad) - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
      c_omega_tmp1_needsGpuEnsureCapa = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#smoothCFS_kernel199#" MW_AT_LINE);
        smoothCFS_kernel199<<<grid, block>>>(i, gpu_cfsDFT);
      }
    } else {
      i = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
      cpu_cfsDFT->size[0] = cpu_cfs->size[0];
      cpu_cfsDFT->size[1] = static_cast<int32_T>(npad);
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsDFT, i, &ud_emlrtRTEI);
      c_omega_tmp1_needsGpuEnsureCapa = true;
    }
    ySize_idx_1 = cpu_cfs->size[0];
    if (cpu_cfs->size[1] > static_cast<int32_T>(npad)) {
      x = static_cast<int32_T>(npad);
    } else {
      x = cpu_cfs->size[1];
    }
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(x - 1, ySize_idx_1 - 1), &grid,
                            &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
    if (c_omega_tmp1_needsGpuEnsureCapa) {
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
    }
    if (*cfs_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_cfs, cpu_cfs);
    }
    *cfs_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel200#" MW_AT_LINE);
      smoothCFS_kernel200<<<grid, block>>>(*gpu_cfs, ySize_idx_1 - 1, x - 1,
                                           gpu_cfsDFT, cpu_cfsDFT->size[0U],
                                           cpu_cfs->size[0U]);
    }
    ySize_idx_1 = cpu_cfsDFT->size[1];
    b_fftPlanHandle =
        acquireCUFFTPlan(1, &ySize_idx_1, &ySize_idx_1, cpu_cfsDFT->size[0], 1,
                         HIPFFT_Z2Z, cpu_cfsDFT->size[0]);
    hipfftExecZ2Z(b_fftPlanHandle, (hipfftDoubleComplex *)&gpu_cfsDFT.data[0],
                 (hipfftDoubleComplex *)&gpu_cfsDFT.data[0], HIPFFT_FORWARD);
  } else {
    hipfftHandle fftPlanHandle;
    ySize_idx_1 = cpu_cfs->size[1];
    fftPlanHandle =
        acquireCUFFTPlan(1, &ySize_idx_1, &ySize_idx_1, cpu_cfs->size[0], 1,
                         HIPFFT_Z2Z, cpu_cfs->size[0]);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
    if (*cfs_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_cfs, cpu_cfs);
    }
    hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&gpu_cfs->data[0],
                 (hipfftDoubleComplex *)&gpu_cfs->data[0], HIPFFT_FORWARD);
    i = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
    cpu_cfsDFT->size[0] = cpu_cfs->size[0];
    cpu_cfsDFT->size[1] = cpu_cfs->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_cfsDFT, i, &og_emlrtRTEI);
    i = cpu_cfs->size[0] * cpu_cfs->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
    *cfs_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel201#" MW_AT_LINE);
      smoothCFS_kernel201<<<grid, block>>>(*gpu_cfs, i, gpu_cfsDFT);
    }
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_Fmat, 2, &ug_emlrtRTEI, true);
  i = cpu_Fmat->size[0] * cpu_Fmat->size[1];
  cpu_Fmat->size[0] = cpu_scales->size[1];
  cpu_Fmat->size[1] = cpu_omega->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_Fmat, i, &pg_emlrtRTEI);
  ySize_idx_1 = cpu_omega->size[1];
  i = cpu_scales->size[1] - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(ySize_idx_1 - 1, i), &grid, &block,
                          2147483647U);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_scales, gpu_scales, !*scales_outdatedOnGpu);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_Fmat, &gpu_Fmat, true);
  if (*scales_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(gpu_scales, cpu_scales);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel178#" MW_AT_LINE);
    smoothCFS_kernel178<<<grid, block>>>(*gpu_scales, gpu_omega, i,
                                         ySize_idx_1 - 1, gpu_Fmat,
                                         cpu_Fmat->size[0U]);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_omega);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_smooth, 2, &vg_emlrtRTEI, true);
  if ((cpu_Fmat->size[0] == cpu_cfsDFT->size[0]) &&
      (cpu_Fmat->size[1] == cpu_cfsDFT->size[1])) {
    nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
    emxInit_creal_T(&b_cpu_Fmat, 2, &xg_emlrtRTEI, true);
    i = b_cpu_Fmat->size[0] * b_cpu_Fmat->size[1];
    b_cpu_Fmat->size[0] = cpu_Fmat->size[0];
    b_cpu_Fmat->size[1] = cpu_Fmat->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(b_cpu_Fmat, i, &xg_emlrtRTEI);
    i = cpu_Fmat->size[0] * cpu_Fmat->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(b_cpu_Fmat, &b_gpu_Fmat, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel179#" MW_AT_LINE);
      smoothCFS_kernel179<<<grid, block>>>(gpu_cfsDFT, gpu_Fmat, i, b_gpu_Fmat);
    }
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_creal_T(b_cpu_Fmat, &b_gpu_Fmat);
    nvtxMarkA("#ifft#" MW_AT_LINE);
    ifft(b_cpu_Fmat, cpu_smooth);
    nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
    emxFree_creal_T(&b_cpu_Fmat);
  } else {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_Fmat, &gpu_Fmat);
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_creal_T(cpu_cfsDFT, &gpu_cfsDFT);
    nvtxMarkA("#binary_expand_op_8#" MW_AT_LINE);
    binary_expand_op_8(cpu_smooth, cpu_Fmat, cpu_cfsDFT);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_Fmat);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfsDFT);
  i2 = cpu_cfs->size[1];
  a = 1.0 / ns;
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_a, 1, &wf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_c, 1, &eg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_expanded, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_expanded, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&c_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv2, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_newIm, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&smooth, 1, &ig_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_smooth, 2, &tg_emlrtRTEI, true);
  if ((cpu_smooth->size[0] == 1) && (static_cast<int32_T>(ns) == 1)) {
    i = cpu_a->size[0];
    cpu_a->size[0] = i2;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_a, i, &wf_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i2 - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_smooth, &gpu_smooth, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_a, &gpu_a, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_creal_T(&gpu_smooth, cpu_smooth);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel180#" MW_AT_LINE);
      smoothCFS_kernel180<<<grid, block>>>(gpu_smooth, i2 - 1, gpu_a,
                                           cpu_smooth->size[0U]);
    }
    i = cpu_c->size[0];
    cpu_c->size[0] = cpu_a->size[0];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_c, i, &ag_emlrtRTEI);
    i = cpu_a->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_c, &gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel181#" MW_AT_LINE);
      smoothCFS_kernel181<<<grid, block>>>(i, gpu_c);
    }
    if (cpu_a->size[0] != 0) {
      if (cpu_a->size[0] == 1) {
        i = cpu_c->size[0];
        cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_c, i, &ag_emlrtRTEI);
        nvtxMarkA("#smoothCFS_kernel183#" MW_AT_LINE);
        smoothCFS_kernel183<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            gpu_a, a, gpu_d, gpu_d1);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_c, &gpu_c, true);
        nvtxMarkA("#smoothCFS_kernel184#" MW_AT_LINE);
        smoothCFS_kernel184<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            gpu_d1, gpu_d, gpu_c);
      } else {
        int32_T OH;
        int8_T threadDims_idx_0;
        x = static_cast<int32_T>(
            std::fmin(32.0, static_cast<real_T>(cpu_a->size[0])));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[0]) +
                                 static_cast<uint32_T>(x)) -
             1.0) /
            static_cast<real_T>(x)));
        threadDims_idx_0 = static_cast<int8_T>(x);
        OH = cpu_a->size[0];
        if (cpu_a->size[0] > 2147483646) {
          ySize_idx_1 = MAX_int32_T;
        } else {
          ySize_idx_1 = cpu_a->size[0] + 1;
        }
        i = b_cpu_expanded->size[0];
        b_cpu_expanded->size[0] = ySize_idx_1 - 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(b_cpu_expanded, i, &cg_emlrtRTEI);
        i = cpu_a->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(b_cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel185#" MW_AT_LINE);
          smoothCFS_kernel185<<<grid, block>>>(i, gpu_expanded);
        }
        x = cpu_a->size[0];
        i = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(b_cpu_y, i, &dg_emlrtRTEI);
        b_cpu_y->data[0] = 1;
        ySize_idx_1 = 1;
        profileLoopStart("b_smoothCFS_loop_1", __LINE__, (x - 2) + 1, "");
        for (int32_T k{0}; k <= x - 2; k++) {
          ySize_idx_1++;
          b_cpu_y->data[k + 1] = ySize_idx_1;
        }
        profileLoopEnd();
        i = cpu_iv1->size[0];
        cpu_iv1->size[0] = b_cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv1, i, &cg_emlrtRTEI);
        i = b_cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(b_cpu_y, &gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, b_cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel186#" MW_AT_LINE);
          smoothCFS_kernel186<<<grid, block>>>(gpu_y, i, gpu_iv1);
        }
        ySize_idx_1 = cpu_iv1->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(ySize_idx_1 - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel187#" MW_AT_LINE);
          smoothCFS_kernel187<<<grid, block>>>(gpu_a, gpu_iv1, ySize_idx_1 - 1,
                                               gpu_expanded);
        }
        i = cpu_c->size[0];
        cpu_c->size[0] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_c, i, &cg_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(
            computeNumIters(OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid,
            &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_c, &gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel188#" MW_AT_LINE);
          smoothCFS_kernel188<<<grid, block>>>(gpu_expanded, a, OH - 1, gpu_c);
        }
      }
    }
    i = cpu_cfs->size[0] * cpu_cfs->size[1];
    cpu_cfs->size[0] = 1;
    cpu_cfs->size[1] = cpu_c->size[0];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_cfs, i, &sg_emlrtRTEI);
    i = cpu_c->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
    if (*cfs_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_cfs, cpu_cfs);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel182#" MW_AT_LINE);
      smoothCFS_kernel182<<<grid, block>>>(gpu_c, i, *gpu_cfs,
                                           cpu_cfs->size[0U]);
    }
    *cfs_outdatedOnGpu = false;
    *cfs_outdatedOnCpu = true;
  } else {
    int32_T b_smooth[2];
    b_smooth[0] = cpu_smooth->size[0];
    i = cpu_cfs->size[0] * cpu_cfs->size[1];
    cpu_cfs->size[0] = cpu_smooth->size[0];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_cfs, i, &sg_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i2 - 1, b_smooth[0] - 1), &grid,
                            &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
    if (*cfs_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_cfs, cpu_cfs);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel189#" MW_AT_LINE);
      smoothCFS_kernel189<<<grid, block>>>(b_smooth[0] - 1, i2 - 1, *gpu_cfs,
                                           cpu_cfs->size[0U]);
    }
    *cfs_outdatedOnGpu = false;
    *cfs_outdatedOnCpu = true;
    if ((cpu_smooth->size[0] != 0) && (i2 != 0) &&
        (static_cast<int32_T>(ns) != 0)) {
      if ((cpu_smooth->size[0] == 1) && (i2 == 1) &&
          (static_cast<int32_T>(ns) == 1)) {
        creal_T cpu_a_data[1];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(ns) - 1),
                                &grid, &block, 2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel190#" MW_AT_LINE);
          smoothCFS_kernel190<<<grid, block>>>(a, static_cast<int32_T>(ns) - 1,
                                               *gpu_a_data);
        }
        i = smooth->size[0];
        smooth->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(smooth, i, &ig_emlrtRTEI);
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(
            hipMemcpy(cpu_a_data, *gpu_a_data, 16UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
        smooth->data[0].re = cpu_smooth->data[0].re * cpu_a_data[0].re -
                             cpu_smooth->data[0].im * cpu_a_data[0].im;
        smooth->data[0].im = cpu_smooth->data[0].re * cpu_a_data[0].im +
                             cpu_smooth->data[0].im * cpu_a_data[0].re;
        i = cpu_cfs->size[0] * cpu_cfs->size[1];
        cpu_cfs->size[0] = 1;
        cpu_cfs->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_cfs, i, &sg_emlrtRTEI);
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_creal_T(cpu_cfs, gpu_cfs);
        cpu_cfs->data[0] = smooth->data[0];
        *cfs_outdatedOnCpu = false;
        *cfs_outdatedOnGpu = true;
      } else {
        int32_T OH;
        boolean_T cfs_needsGpuEnsureCapacity;
        boolean_T rows_needsGpuEnsureCapacity;
        OH = cpu_smooth->size[0];
        if (static_cast<int32_T>(ns) > MAX_int32_T - cpu_smooth->size[0]) {
          ySize_idx_1 = MAX_int32_T;
        } else {
          ySize_idx_1 = cpu_smooth->size[0] + static_cast<int32_T>(ns);
        }
        i = cpu_expanded->size[0] * cpu_expanded->size[1];
        cpu_expanded->size[0] = ySize_idx_1 - 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_expanded, i, &cg_emlrtRTEI);
        if (i2 > 2147483646) {
          ySize_idx_1 = MAX_int32_T;
        } else {
          ySize_idx_1 = i2 + 1;
        }
        i = cpu_expanded->size[0] * cpu_expanded->size[1];
        cpu_expanded->size[1] = ySize_idx_1 - 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_expanded, i, &cg_emlrtRTEI);
        i = ((cpu_smooth->size[0] + static_cast<int32_T>(ns)) - 1) * i2 - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_expanded, &b_gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel191#" MW_AT_LINE);
          smoothCFS_kernel191<<<grid, block>>>(i, b_gpu_expanded);
        }
        x = cpu_smooth->size[0];
        i = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = cpu_smooth->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_y, i, &dg_emlrtRTEI);
        cpu_y->data[0] = 1;
        ySize_idx_1 = 1;
        profileLoopStart("b_smoothCFS_loop_0", __LINE__, (x - 2) + 1, "");
        for (int32_T k{0}; k <= x - 2; k++) {
          ySize_idx_1++;
          cpu_y->data[k + 1] = ySize_idx_1;
        }
        profileLoopEnd();
        i = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, i, &cg_emlrtRTEI);
        i = cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &b_gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel192#" MW_AT_LINE);
          smoothCFS_kernel192<<<grid, block>>>(
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(static_cast<int32_T>(ns)) - 1.0) / 2.0)),
              b_gpu_y, i, gpu_iv);
        }
        i = c_cpu_y->size[0] * c_cpu_y->size[1];
        c_cpu_y->size[0] = 1;
        c_cpu_y->size[1] = i2;
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(c_cpu_y, i, &dg_emlrtRTEI);
        c_cpu_y->data[0] = 1;
        ySize_idx_1 = 1;
        profileLoopStart("b_smoothCFS_loop_2", __LINE__, (i2 - 2) + 1, "");
        for (int32_T k{0}; k <= i2 - 2; k++) {
          ySize_idx_1++;
          c_cpu_y->data[k + 1] = ySize_idx_1;
        }
        profileLoopEnd();
        i = cpu_iv2->size[0];
        cpu_iv2->size[0] = c_cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv2, i, &cg_emlrtRTEI);
        i = c_cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(c_cpu_y, &c_gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv2, &gpu_iv2, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&c_gpu_y, c_cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel193#" MW_AT_LINE);
          smoothCFS_kernel193<<<grid, block>>>(c_gpu_y, i, gpu_iv2);
        }
        i = b_cpu_smooth->size[0] * b_cpu_smooth->size[1];
        b_cpu_smooth->size[0] = cpu_smooth->size[0];
        b_cpu_smooth->size[1] = i2;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(b_cpu_smooth, i, &tg_emlrtRTEI);
        i = cpu_smooth->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i2 - 1, i), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_smooth, &gpu_smooth, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(b_cpu_smooth, &b_gpu_smooth, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_smooth, cpu_smooth);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel194#" MW_AT_LINE);
          smoothCFS_kernel194<<<grid, block>>>(
              gpu_smooth, i, i2 - 1, b_gpu_smooth, b_cpu_smooth->size[0U],
              cpu_smooth->size[0U]);
        }
        b_smooth[0] = cpu_iv->size[0];
        b_smooth[1] = cpu_iv2->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(b_smooth[1] - 1, b_smooth[0] - 1), &grid, &block,
            2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel195#" MW_AT_LINE);
          smoothCFS_kernel195<<<grid, block>>>(
              b_gpu_smooth, b_smooth[0], gpu_iv2, gpu_iv, b_smooth[0] - 1,
              b_smooth[1] - 1, b_gpu_expanded, cpu_expanded->size[0U]);
        }
        if (static_cast<int32_T>(ns) - 1 < 0) {
          x = 0;
        } else {
          x = static_cast<int32_T>(ns);
        }
        c_omega_tmp1_needsGpuEnsureCapa = false;
        i = cpu_rows->size[0] * cpu_rows->size[1];
        cpu_rows->size[0] = 1;
        cpu_rows->size[1] = x;
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_rows, i, &dg_emlrtRTEI);
        rows_needsGpuEnsureCapacity = true;
        if (x > 0) {
          cpu_rows->data[0] = 0;
          c_omega_tmp1_needsGpuEnsureCapa = true;
          ySize_idx_1 = 0;
          profileLoopStart("b_smoothCFS_loop_3", __LINE__, (x - 2) + 1, "");
          for (int32_T k{0}; k <= x - 2; k++) {
            ySize_idx_1++;
            cpu_rows->data[k + 1] = ySize_idx_1;
          }
          profileLoopEnd();
        }
        i = cpu_cfs->size[0] * cpu_cfs->size[1];
        cpu_cfs->size[0] = cpu_smooth->size[0];
        cpu_cfs->size[1] = i2;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_cfs, i, &cg_emlrtRTEI);
        cfs_needsGpuEnsureCapacity = true;
        ySize_idx_1 = static_cast<int32_T>(ns);
        profileLoopStart("b_smoothCFS_loop_4", __LINE__, (i2 - 1) + 1, "");
        for (offsetH = 0; offsetH < i2; offsetH++) {
          profileLoopStart("b_smoothCFS_loop_5", __LINE__, (OH - 1) + 1, "");
          for (int32_T k{0}; k < OH; k++) {
            real_T cv_im;
            boolean_T newIm_outdatedOnCpu;
            i = cpu_newIm->size[0];
            cpu_newIm->size[0] = cpu_rows->size[1];
            nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
            emxEnsureCapacity_creal_T(cpu_newIm, i, &cg_emlrtRTEI);
            i = cpu_rows->size[1] - 1;
            nvtxMarkA("#computeNumIters#" MW_AT_LINE);
            mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                    2147483647U);
            if (rows_needsGpuEnsureCapacity) {
              nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
              gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows,
                                           !c_omega_tmp1_needsGpuEnsureCapa);
            }
            rows_needsGpuEnsureCapacity = false;
            nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_creal_T(cpu_newIm, &gpu_newIm, true);
            if (c_omega_tmp1_needsGpuEnsureCapa) {
              nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
              gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
            }
            c_omega_tmp1_needsGpuEnsureCapa = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              nvtxMarkA("#smoothCFS_kernel196#" MW_AT_LINE);
              smoothCFS_kernel196<<<grid, block>>>(offsetH, b_gpu_expanded, k,
                                                   gpu_rows, i, gpu_newIm,
                                                   cpu_expanded->size[0U]);
            }
            newIm_outdatedOnCpu = true;
            npad = 0.0;
            cv_im = 0.0;
            profileLoopStart("b_smoothCFS_loop_6", __LINE__,
                             (ySize_idx_1 - 1) + 1, "");
            for (x = 0; x < ySize_idx_1; x++) {
              if (newIm_outdatedOnCpu) {
                nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
                gpuEmxMemcpyGpuToCpu_creal_T(cpu_newIm, &gpu_newIm);
              }
              npad += cpu_newIm->data[x].re * a;
              newIm_outdatedOnCpu = false;
              cv_im += cpu_newIm->data[x].im * a;
            }
            profileLoopEnd();
            if (cfs_needsGpuEnsureCapacity) {
              nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
              gpuEmxEnsureCapacity_creal_T(cpu_cfs, gpu_cfs, true);
            }
            nvtxMarkA("#smoothCFS_kernel197#" MW_AT_LINE);
            smoothCFS_kernel197<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                npad, offsetH, k, *gpu_cfs, cpu_cfs->size[0U]);
            cfs_needsGpuEnsureCapacity = false;
            nvtxMarkA("#smoothCFS_kernel198#" MW_AT_LINE);
            smoothCFS_kernel198<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                cv_im, offsetH, k, *gpu_cfs, cpu_cfs->size[0U]);
          }
          profileLoopEnd();
        }
        profileLoopEnd();
      }
    }
  }
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_smooth);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&smooth);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_newIm);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv2);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&c_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_expanded);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_expanded);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_c);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_a);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_smooth);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_omega_tmp1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_omega_tmp2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_omega);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfsDFT);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_Fmat);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_smooth);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_Fmat);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_c);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv2);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_smooth);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_a_data), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_d), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_d1), __FILE__, __LINE__);
  *scales_outdatedOnGpu = false;
  nvtxRangePop();
}

//
//
void smoothCFS(emxArray_real_T *cpu_cfs, boolean_T *cfs_outdatedOnCpu,
               emxArray_real_T *gpu_cfs, boolean_T *cfs_outdatedOnGpu,
               emxArray_real_T *cpu_scales, emxArray_real_T *gpu_scales,
               boolean_T *scales_outdatedOnGpu, real_T ns)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_Fmat;
  emxArray_creal_T gpu_cfsDFT;
  emxArray_creal_T gpu_r;
  emxArray_creal_T *b_cpu_Fmat;
  emxArray_creal_T *cpu_cfsDFT;
  emxArray_creal_T *cpu_r;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T c_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_iv2;
  emxArray_int32_T gpu_omega_tmp1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *c_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_iv2;
  emxArray_int32_T *cpu_omega_tmp1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T b_gpu_c;
  emxArray_real_T b_gpu_expanded;
  emxArray_real_T b_gpu_smooth;
  emxArray_real_T c_gpu_smooth;
  emxArray_real_T gpu_Fmat;
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_c;
  emxArray_real_T gpu_expanded;
  emxArray_real_T gpu_omega;
  emxArray_real_T gpu_omega_tmp2;
  emxArray_real_T gpu_smooth;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *b_cpu_expanded;
  emxArray_real_T *b_cpu_smooth;
  emxArray_real_T *c_cpu_smooth;
  emxArray_real_T *cpu_Fmat;
  emxArray_real_T *cpu_a;
  emxArray_real_T *cpu_c;
  emxArray_real_T *cpu_expanded;
  emxArray_real_T *cpu_omega;
  emxArray_real_T *cpu_omega_tmp2;
  emxArray_real_T *cpu_smooth;
  real_T(*gpu_a_data)[1];
  real_T npad;
  int32_T N;
  int32_T i2;
  int32_T k;
  int32_T midpoint;
  int32_T offsetH;
  boolean_T c_omega_tmp1_needsGpuEnsureCapa;
  boolean_T c_outdatedOnCpu;
  boolean_T c_outdatedOnGpu;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#smoothCFS#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a_data, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_smooth);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_smooth);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv2);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_c);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_a);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_smooth);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_Fmat);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_Fmat);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfsDFT);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_omega);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_omega_tmp2);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_omega_tmp1);
  c_outdatedOnCpu = false;
  c_outdatedOnGpu = false;
  c_omega_tmp1_needsGpuEnsureCapa = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  npad = std::frexp(static_cast<real_T>(cpu_cfs->size[1]), &N);
  if (npad == 0.5) {
    N--;
  }
  nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
  npad = rt_powd_snf(2.0, static_cast<real_T>(N));
  N = static_cast<int32_T>(std::trunc(npad / 2.0));
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_omega_tmp1, 2, &lg_emlrtRTEI, true);
  if (N < 1) {
    cpu_omega_tmp1->size[0] = 1;
    cpu_omega_tmp1->size[1] = 0;
  } else {
    k = cpu_omega_tmp1->size[0] * cpu_omega_tmp1->size[1];
    cpu_omega_tmp1->size[0] = 1;
    cpu_omega_tmp1->size[1] = N;
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_omega_tmp1, k, &lg_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(N - 1), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_omega_tmp1, &gpu_omega_tmp1, true);
    c_omega_tmp1_needsGpuEnsureCapa = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel172#" MW_AT_LINE);
      smoothCFS_kernel172<<<grid, block>>>(N - 1, gpu_omega_tmp1);
    }
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_omega_tmp2, 2, &mg_emlrtRTEI, true);
  k = cpu_omega_tmp2->size[0] * cpu_omega_tmp2->size[1];
  cpu_omega_tmp2->size[0] = 1;
  cpu_omega_tmp2->size[1] = cpu_omega_tmp1->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_omega_tmp2, k, &mg_emlrtRTEI);
  k = cpu_omega_tmp1->size[1] - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  if (c_omega_tmp1_needsGpuEnsureCapa) {
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_omega_tmp1, &gpu_omega_tmp1, true);
  }
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_omega_tmp2, &gpu_omega_tmp2, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel142#" MW_AT_LINE);
    smoothCFS_kernel142<<<grid, block>>>(6.2831853071795862 / npad,
                                         gpu_omega_tmp1, k, gpu_omega_tmp2);
  }
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_omega_tmp1);
  N = static_cast<int32_T>(std::trunc((npad - 1.0) / 2.0));
  if (N < 1) {
    midpoint = 0;
    offsetH = 1;
    i2 = -1;
  } else {
    midpoint = N - 1;
    offsetH = -1;
    i2 = 0;
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_omega, 2, &ng_emlrtRTEI, true);
  k = cpu_omega->size[0] * cpu_omega->size[1];
  cpu_omega->size[0] = 1;
  nvtxMarkA("#div_s32#" MW_AT_LINE);
  cpu_omega->size[1] =
      (cpu_omega_tmp2->size[1] + div_s32(i2 - midpoint, offsetH)) + 2;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_omega, k, &ng_emlrtRTEI);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_omega, &gpu_omega, true);
  nvtxMarkA("#smoothCFS_kernel143#" MW_AT_LINE);
  smoothCFS_kernel143<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_omega);
  N = cpu_omega_tmp2->size[1];
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(N - 1), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel144#" MW_AT_LINE);
    smoothCFS_kernel144<<<grid, block>>>(gpu_omega_tmp2, N - 1, gpu_omega);
  }
  nvtxMarkA("#div_s32#" MW_AT_LINE);
  N = div_s32(i2 - midpoint, offsetH);
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(N), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel145#" MW_AT_LINE);
    smoothCFS_kernel145<<<grid, block>>>(offsetH, midpoint, gpu_omega_tmp2, N,
                                         gpu_omega, cpu_omega_tmp2->size[1U]);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_omega_tmp2);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfsDFT, 2, &og_emlrtRTEI, true);
  if ((cpu_cfs->size[0] == 0) || (cpu_cfs->size[1] == 0) ||
      (static_cast<int32_T>(npad) == 0)) {
    k = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
    cpu_cfsDFT->size[0] = cpu_cfs->size[0];
    cpu_cfsDFT->size[1] = static_cast<int32_T>(npad);
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_cfsDFT, k, &og_emlrtRTEI);
    k = cpu_cfs->size[0] * static_cast<int32_T>(npad) - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel146#" MW_AT_LINE);
      smoothCFS_kernel146<<<grid, block>>>(k, gpu_cfsDFT);
    }
  } else if (cpu_cfs->size[1] != static_cast<int32_T>(npad)) {
    hipfftHandle c_fftPlanHandle;
    if (cpu_cfs->size[1] < static_cast<int32_T>(npad)) {
      k = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
      cpu_cfsDFT->size[0] = cpu_cfs->size[0];
      cpu_cfsDFT->size[1] = static_cast<int32_T>(npad);
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsDFT, k, &og_emlrtRTEI);
      k = cpu_cfs->size[0] * static_cast<int32_T>(npad) - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
      c_omega_tmp1_needsGpuEnsureCapa = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#smoothCFS_kernel167#" MW_AT_LINE);
        smoothCFS_kernel167<<<grid, block>>>(k, gpu_cfsDFT);
      }
    } else {
      k = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
      cpu_cfsDFT->size[0] = cpu_cfs->size[0];
      cpu_cfsDFT->size[1] = static_cast<int32_T>(npad);
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsDFT, k, &ud_emlrtRTEI);
      c_omega_tmp1_needsGpuEnsureCapa = true;
    }
    N = cpu_cfs->size[0];
    if (cpu_cfs->size[1] > static_cast<int32_T>(npad)) {
      midpoint = static_cast<int32_T>(npad);
    } else {
      midpoint = cpu_cfs->size[1];
    }
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(midpoint - 1, N - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
    if (c_omega_tmp1_needsGpuEnsureCapa) {
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
    }
    if (*cfs_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_cfs, cpu_cfs);
    }
    *cfs_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel168#" MW_AT_LINE);
      smoothCFS_kernel168<<<grid, block>>>(*gpu_cfs, N - 1, midpoint - 1,
                                           gpu_cfsDFT, cpu_cfsDFT->size[0U],
                                           cpu_cfs->size[0U]);
    }
    N = cpu_cfsDFT->size[1];
    c_fftPlanHandle = acquireCUFFTPlan(1, &N, &N, cpu_cfsDFT->size[0], 1,
                                       HIPFFT_Z2Z, cpu_cfsDFT->size[0]);
    hipfftExecZ2Z(c_fftPlanHandle, (hipfftDoubleComplex *)&gpu_cfsDFT.data[0],
                 (hipfftDoubleComplex *)&gpu_cfsDFT.data[0], HIPFFT_FORWARD);
  } else {
    midpoint = cpu_cfs->size[0] * cpu_cfs->size[1];
    if (midpoint -
            (static_cast<int32_T>(static_cast<uint32_T>(midpoint) >> 1) << 1) ==
        1) {
      hipfftHandle fftPlanHandle;
      k = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
      cpu_cfsDFT->size[0] = cpu_cfs->size[0];
      cpu_cfsDFT->size[1] = cpu_cfs->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsDFT, k, &og_emlrtRTEI);
      k = cpu_cfs->size[0] * cpu_cfs->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
      if (*cfs_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_cfs, cpu_cfs);
      }
      *cfs_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#smoothCFS_kernel169#" MW_AT_LINE);
        smoothCFS_kernel169<<<grid, block>>>(*gpu_cfs, k, gpu_cfsDFT);
      }
      N = cpu_cfsDFT->size[1];
      fftPlanHandle = acquireCUFFTPlan(1, &N, &N, cpu_cfsDFT->size[0], 1,
                                       HIPFFT_Z2Z, cpu_cfsDFT->size[0]);
      hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&gpu_cfsDFT.data[0],
                   (hipfftDoubleComplex *)&gpu_cfsDFT.data[0], HIPFFT_FORWARD);
    } else {
      hipfftHandle b_fftPlanHandle;
      uint32_T xSize[2];
      N = cpu_cfs->size[1];
      profileLoopStart("smoothCFS_loop_0", __LINE__, 1 + 1, "");
      for (k = 0; k < 2; k++) {
        xSize[k] = static_cast<uint32_T>(cpu_cfs->size[k]);
      }
      profileLoopEnd();
      k = cpu_cfsDFT->size[0] * cpu_cfsDFT->size[1];
      cpu_cfsDFT->size[0] = static_cast<int32_T>(xSize[0]);
      cpu_cfsDFT->size[1] = static_cast<int32_T>(xSize[1]);
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsDFT, k, &ud_emlrtRTEI);
      b_fftPlanHandle = acquireCUFFTPlan(1, &N, &N, cpu_cfs->size[0], 1,
                                         HIPFFT_D2Z, cpu_cfs->size[0]);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfsDFT, &gpu_cfsDFT, true);
      if (*cfs_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_cfs, cpu_cfs);
      }
      *cfs_outdatedOnGpu = false;
      hipfftExecD2Z(b_fftPlanHandle, (hipfftDoubleReal *)&gpu_cfs->data[0],
                   (hipfftDoubleComplex *)&gpu_cfsDFT.data[0]);
      midpoint =
          static_cast<int32_T>(
              static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) / 2.0) -
          2;
      if ((cpu_cfsDFT->size[0] == 1) || (cpu_cfsDFT->size[1] == 1)) {
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(midpoint), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel170#" MW_AT_LINE);
          smoothCFS_kernel170<<<grid, block>>>(static_cast<int32_T>(xSize[1]),
                                               midpoint, gpu_cfsDFT);
        }
      } else {
        k = cpu_cfsDFT->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(midpoint, k), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel171#" MW_AT_LINE);
          smoothCFS_kernel171<<<grid, block>>>(static_cast<int32_T>(xSize[1]),
                                               k, midpoint, gpu_cfsDFT,
                                               cpu_cfsDFT->size[0U]);
        }
      }
    }
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_Fmat, 2, &ug_emlrtRTEI, true);
  k = cpu_Fmat->size[0] * cpu_Fmat->size[1];
  cpu_Fmat->size[0] = cpu_scales->size[1];
  cpu_Fmat->size[1] = cpu_omega->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_Fmat, k, &pg_emlrtRTEI);
  N = cpu_omega->size[1];
  k = cpu_scales->size[1] - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(N - 1, k), &grid, &block,
                          2147483647U);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_scales, gpu_scales, !*scales_outdatedOnGpu);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_Fmat, &gpu_Fmat, true);
  if (*scales_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(gpu_scales, cpu_scales);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel147#" MW_AT_LINE);
    smoothCFS_kernel147<<<grid, block>>>(*gpu_scales, gpu_omega, k, N - 1,
                                         gpu_Fmat, cpu_Fmat->size[0U]);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_omega);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r, 2, &wg_emlrtRTEI, true);
  if ((cpu_Fmat->size[0] == cpu_cfsDFT->size[0]) &&
      (cpu_Fmat->size[1] == cpu_cfsDFT->size[1])) {
    nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
    emxInit_creal_T(&b_cpu_Fmat, 2, &qg_emlrtRTEI, true);
    k = b_cpu_Fmat->size[0] * b_cpu_Fmat->size[1];
    b_cpu_Fmat->size[0] = cpu_Fmat->size[0];
    b_cpu_Fmat->size[1] = cpu_Fmat->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(b_cpu_Fmat, k, &qg_emlrtRTEI);
    k = cpu_Fmat->size[0] * cpu_Fmat->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(b_cpu_Fmat, &b_gpu_Fmat, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel148#" MW_AT_LINE);
      smoothCFS_kernel148<<<grid, block>>>(gpu_cfsDFT, gpu_Fmat, k, b_gpu_Fmat);
    }
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_creal_T(b_cpu_Fmat, &b_gpu_Fmat);
    nvtxMarkA("#ifft#" MW_AT_LINE);
    ifft(b_cpu_Fmat, cpu_r);
    nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
    emxFree_creal_T(&b_cpu_Fmat);
  } else {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_Fmat, &gpu_Fmat);
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_creal_T(cpu_cfsDFT, &gpu_cfsDFT);
    nvtxMarkA("#binary_expand_op_8#" MW_AT_LINE);
    binary_expand_op_8(cpu_r, cpu_Fmat, cpu_cfsDFT);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_Fmat);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfsDFT);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_smooth, 2, &vg_emlrtRTEI, true);
  k = cpu_smooth->size[0] * cpu_smooth->size[1];
  cpu_smooth->size[0] = cpu_r->size[0];
  cpu_smooth->size[1] = cpu_r->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_smooth, k, &rg_emlrtRTEI);
  k = cpu_r->size[0] * cpu_r->size[1] - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_creal_T(cpu_r, &gpu_r, false);
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_smooth, &gpu_smooth, true);
  nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
  gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r, cpu_r);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#smoothCFS_kernel149#" MW_AT_LINE);
    smoothCFS_kernel149<<<grid, block>>>(gpu_r, k, gpu_smooth);
  }
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r);
  i2 = cpu_cfs->size[1];
  npad = 1.0 / ns;
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_a, 1, &wf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_c, 1, &gg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_c, 1, &eg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_expanded, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_expanded, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&c_cpu_y, 2, &fg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv2, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_smooth, 2, &tg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_smooth, 2, &tg_emlrtRTEI, true);
  if ((cpu_smooth->size[0] == 1) && (static_cast<int32_T>(ns) == 1)) {
    k = cpu_a->size[0];
    cpu_a->size[0] = i2;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_a, k, &wf_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i2 - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel150#" MW_AT_LINE);
      smoothCFS_kernel150<<<grid, block>>>(gpu_smooth, i2 - 1, gpu_a,
                                           cpu_smooth->size[0U]);
    }
    k = b_cpu_c->size[0];
    b_cpu_c->size[0] = cpu_a->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b_cpu_c, k, &ag_emlrtRTEI);
    k = cpu_a->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(b_cpu_c, &gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel151#" MW_AT_LINE);
      smoothCFS_kernel151<<<grid, block>>>(k, gpu_c);
    }
    if (cpu_a->size[0] != 0) {
      if (cpu_a->size[0] == 1) {
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_c, k, &ag_emlrtRTEI);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_c, &gpu_c, true);
        nvtxMarkA("#smoothCFS_kernel153#" MW_AT_LINE);
        smoothCFS_kernel153<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            npad, gpu_a, gpu_c);
      } else {
        int32_T OH;
        int8_T threadDims[3];
        midpoint = static_cast<int32_T>(
            std::fmin(32.0, static_cast<real_T>(cpu_a->size[0])));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[0]) +
                                 static_cast<uint32_T>(midpoint)) -
             1.0) /
            static_cast<real_T>(midpoint)));
        threadDims[0] = static_cast<int8_T>(midpoint);
        OH = cpu_a->size[0];
        if (cpu_a->size[0] > 2147483646) {
          N = MAX_int32_T;
        } else {
          N = cpu_a->size[0] + 1;
        }
        k = b_cpu_expanded->size[0];
        b_cpu_expanded->size[0] = N - 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_expanded, k, &cg_emlrtRTEI);
        k = cpu_a->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel154#" MW_AT_LINE);
          smoothCFS_kernel154<<<grid, block>>>(k, gpu_expanded);
        }
        midpoint = cpu_a->size[0];
        k = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(b_cpu_y, k, &dg_emlrtRTEI);
        b_cpu_y->data[0] = 1;
        N = 1;
        profileLoopStart("smoothCFS_loop_1", __LINE__, (midpoint - 2) + 1, "");
        for (k = 0; k <= midpoint - 2; k++) {
          N++;
          b_cpu_y->data[k + 1] = N;
        }
        profileLoopEnd();
        k = cpu_iv1->size[0];
        cpu_iv1->size[0] = b_cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv1, k, &cg_emlrtRTEI);
        k = b_cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(b_cpu_y, &gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, b_cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel155#" MW_AT_LINE);
          smoothCFS_kernel155<<<grid, block>>>(gpu_y, k, gpu_iv1);
        }
        N = cpu_iv1->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(N - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel156#" MW_AT_LINE);
          smoothCFS_kernel156<<<grid, block>>>(gpu_a, gpu_iv1, N - 1,
                                               gpu_expanded);
        }
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_c, k, &cg_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(
            computeNumIters(OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims[0]), 1U, 1U), &grid, &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_c, &gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel157#" MW_AT_LINE);
          smoothCFS_kernel157<<<grid, block>>>(npad, gpu_expanded, OH - 1,
                                               gpu_c);
        }
      }
    }
    k = cpu_cfs->size[0] * cpu_cfs->size[1];
    cpu_cfs->size[0] = 1;
    cpu_cfs->size[1] = b_cpu_c->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_cfs, k, &sg_emlrtRTEI);
    k = b_cpu_c->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
    if (*cfs_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_cfs, cpu_cfs);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel152#" MW_AT_LINE);
      smoothCFS_kernel152<<<grid, block>>>(gpu_c, k, *gpu_cfs,
                                           cpu_cfs->size[0U]);
    }
    *cfs_outdatedOnGpu = false;
    *cfs_outdatedOnCpu = true;
  } else {
    int32_T smooth[2];
    smooth[0] = cpu_smooth->size[0];
    k = cpu_cfs->size[0] * cpu_cfs->size[1];
    cpu_cfs->size[0] = cpu_smooth->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_cfs, k, &sg_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i2 - 1, smooth[0] - 1), &grid,
                            &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_cfs, gpu_cfs, !*cfs_outdatedOnGpu);
    if (*cfs_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_cfs, cpu_cfs);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#smoothCFS_kernel158#" MW_AT_LINE);
      smoothCFS_kernel158<<<grid, block>>>(smooth[0] - 1, i2 - 1, *gpu_cfs,
                                           cpu_cfs->size[0U]);
    }
    *cfs_outdatedOnGpu = false;
    *cfs_outdatedOnCpu = true;
    if ((cpu_smooth->size[0] != 0) && (i2 != 0) &&
        (static_cast<int32_T>(ns) != 0)) {
      if ((cpu_smooth->size[0] == 1) && (i2 == 1) &&
          (static_cast<int32_T>(ns) == 1)) {
        real_T cpu_a_data[1];
        k = b_cpu_smooth->size[0] * b_cpu_smooth->size[1];
        b_cpu_smooth->size[0] = 1;
        b_cpu_smooth->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_smooth, k, &tg_emlrtRTEI);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_smooth, &b_gpu_smooth, true);
        nvtxMarkA("#smoothCFS_kernel159#" MW_AT_LINE);
        smoothCFS_kernel159<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            gpu_smooth, b_gpu_smooth);
        c_omega_tmp1_needsGpuEnsureCapa = false;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(ns) - 1),
                                &grid, &block, 2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel160#" MW_AT_LINE);
          smoothCFS_kernel160<<<grid, block>>>(
              npad, static_cast<int32_T>(ns) - 1, *gpu_a_data);
        }
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(
            hipMemcpy(cpu_a_data, *gpu_a_data, 8UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
        nvtxMarkA("#mtimes#" MW_AT_LINE);
        ::coder::internal::blas::mtimes(
            b_cpu_smooth, &b_gpu_smooth, &c_omega_tmp1_needsGpuEnsureCapa,
            cpu_a_data, cpu_c, &c_outdatedOnCpu, &b_gpu_c, &c_outdatedOnGpu);
        k = cpu_cfs->size[0] * cpu_cfs->size[1];
        cpu_cfs->size[0] = 1;
        cpu_cfs->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_cfs, k, &sg_emlrtRTEI);
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_cfs, gpu_cfs);
        if (c_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_c, &b_gpu_c);
        }
        cpu_cfs->data[0] = cpu_c->data[0];
        *cfs_outdatedOnCpu = false;
        *cfs_outdatedOnGpu = true;
      } else {
        int32_T OH;
        uint32_T blockDims[3];
        int8_T threadDims[3];
        midpoint = static_cast<int32_T>(
            std::fmin(32.0, static_cast<real_T>(cpu_smooth->size[0])));
        N = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(i2)));
        blockDims[0] = static_cast<uint32_T>(std::floor(
            static_cast<real_T>((static_cast<uint32_T>(cpu_smooth->size[0]) +
                                 static_cast<uint32_T>(midpoint)) -
                                1U) /
            static_cast<real_T>(midpoint)));
        blockDims[1] = static_cast<uint32_T>(std::floor(
            static_cast<real_T>(
                (static_cast<uint32_T>(i2) + static_cast<uint32_T>(N)) - 1U) /
            static_cast<real_T>(N)));
        threadDims[0] = static_cast<int8_T>(midpoint);
        threadDims[1] = static_cast<int8_T>(N);
        OH = cpu_smooth->size[0];
        if (static_cast<int32_T>(ns) > MAX_int32_T - cpu_smooth->size[0]) {
          N = MAX_int32_T;
        } else {
          N = cpu_smooth->size[0] + static_cast<int32_T>(ns);
        }
        k = cpu_expanded->size[0] * cpu_expanded->size[1];
        cpu_expanded->size[0] = N - 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_expanded, k, &cg_emlrtRTEI);
        if (i2 > 2147483646) {
          N = MAX_int32_T;
        } else {
          N = i2 + 1;
        }
        k = cpu_expanded->size[0] * cpu_expanded->size[1];
        cpu_expanded->size[1] = N - 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_expanded, k, &cg_emlrtRTEI);
        k = ((cpu_smooth->size[0] + static_cast<int32_T>(ns)) - 1) * i2 - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_expanded, &b_gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel161#" MW_AT_LINE);
          smoothCFS_kernel161<<<grid, block>>>(k, b_gpu_expanded);
        }
        midpoint = cpu_smooth->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = cpu_smooth->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_y, k, &dg_emlrtRTEI);
        cpu_y->data[0] = 1;
        N = 1;
        profileLoopStart("smoothCFS_loop_2", __LINE__, (midpoint - 2) + 1, "");
        for (k = 0; k <= midpoint - 2; k++) {
          N++;
          cpu_y->data[k + 1] = N;
        }
        profileLoopEnd();
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, k, &cg_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &b_gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel162#" MW_AT_LINE);
          smoothCFS_kernel162<<<grid, block>>>(
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(static_cast<int32_T>(ns)) - 1.0) / 2.0)),
              b_gpu_y, k, gpu_iv);
        }
        k = c_cpu_y->size[0] * c_cpu_y->size[1];
        c_cpu_y->size[0] = 1;
        c_cpu_y->size[1] = i2;
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(c_cpu_y, k, &dg_emlrtRTEI);
        c_cpu_y->data[0] = 1;
        N = 1;
        profileLoopStart("smoothCFS_loop_3", __LINE__, (i2 - 2) + 1, "");
        for (k = 0; k <= i2 - 2; k++) {
          N++;
          c_cpu_y->data[k + 1] = N;
        }
        profileLoopEnd();
        k = cpu_iv2->size[0];
        cpu_iv2->size[0] = c_cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv2, k, &cg_emlrtRTEI);
        k = c_cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(c_cpu_y, &c_gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv2, &gpu_iv2, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&c_gpu_y, c_cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel163#" MW_AT_LINE);
          smoothCFS_kernel163<<<grid, block>>>(c_gpu_y, k, gpu_iv2);
        }
        k = c_cpu_smooth->size[0] * c_cpu_smooth->size[1];
        c_cpu_smooth->size[0] = cpu_smooth->size[0];
        c_cpu_smooth->size[1] = i2;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_cpu_smooth, k, &tg_emlrtRTEI);
        k = cpu_smooth->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i2 - 1, k), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(c_cpu_smooth, &c_gpu_smooth, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel164#" MW_AT_LINE);
          smoothCFS_kernel164<<<grid, block>>>(
              gpu_smooth, k, i2 - 1, c_gpu_smooth, c_cpu_smooth->size[0U],
              cpu_smooth->size[0U]);
        }
        smooth[0] = cpu_iv->size[0];
        smooth[1] = cpu_iv2->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(smooth[1] - 1, smooth[0] - 1),
                                &grid, &block, 2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel165#" MW_AT_LINE);
          smoothCFS_kernel165<<<grid, block>>>(
              c_gpu_smooth, smooth[0], gpu_iv2, gpu_iv, smooth[0] - 1,
              smooth[1] - 1, b_gpu_expanded, cpu_expanded->size[0U]);
        }
        if (static_cast<int32_T>(ns) - 1 < 0) {
          midpoint = 0;
        } else {
          midpoint = static_cast<int32_T>(ns);
        }
        c_omega_tmp1_needsGpuEnsureCapa = false;
        k = cpu_rows->size[0] * cpu_rows->size[1];
        cpu_rows->size[0] = 1;
        cpu_rows->size[1] = midpoint;
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_rows, k, &dg_emlrtRTEI);
        if (midpoint > 0) {
          cpu_rows->data[0] = 0;
          c_omega_tmp1_needsGpuEnsureCapa = true;
          N = 0;
          profileLoopStart("smoothCFS_loop_4", __LINE__, (midpoint - 2) + 1,
                           "");
          for (k = 0; k <= midpoint - 2; k++) {
            N++;
            cpu_rows->data[k + 1] = N;
          }
          profileLoopEnd();
        }
        k = cpu_cfs->size[0] * cpu_cfs->size[1];
        cpu_cfs->size[0] = cpu_smooth->size[0];
        cpu_cfs->size[1] = i2;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_cfs, k, &cg_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(computeNumIters(i2 - 1, OH - 1),
                                dim3(blockDims[0], blockDims[1], 1U),
                                dim3(static_cast<uint32_T>(threadDims[0]),
                                     static_cast<uint32_T>(threadDims[1]), 1U),
                                &grid, &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows,
                                     !c_omega_tmp1_needsGpuEnsureCapa);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_cfs, gpu_cfs, true);
        if (c_omega_tmp1_needsGpuEnsureCapa) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#smoothCFS_kernel166#" MW_AT_LINE);
          smoothCFS_kernel166<<<grid, block>>>(
              npad, b_gpu_expanded, gpu_rows, ns, OH - 1, i2 - 1, *gpu_cfs,
              cpu_expanded->size[0U], cpu_cfs->size[0U]);
        }
      }
    }
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_smooth);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_smooth);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv2);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&c_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_expanded);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_expanded);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_c);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_c);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_a);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_smooth);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_omega_tmp1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_omega_tmp2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_omega);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfsDFT);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_Fmat);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_Fmat);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_smooth);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_c);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv2);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_smooth);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_smooth);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_a_data), __FILE__, __LINE__);
  *scales_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (smoothCFS.cu)

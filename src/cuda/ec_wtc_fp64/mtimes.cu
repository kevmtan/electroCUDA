//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mtimes.cu
//
// Code generation for function 'mtimes'
//

// Include files
#include "mtimes.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Variable Definitions
static emlrtRTEInfo
    xd_emlrtRTEI{
        158,      // lineNo
        24,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo
    yd_emlrtRTEI{
        140,      // lineNo
        5,        // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

// Function Declarations
static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line);

// Function Definitions
static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line)
{
  const char *errName;
  const char *errString;
  nvtxRangePushA("#fcn#cublasCheck#" MW_AT_LOCATION);
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    nvtxMarkA("#raiseCudaError#" MW_AT_LINE);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
  nvtxRangePop();
}

//
//
namespace coder {
namespace internal {
namespace blas {
void mtimes(emxArray_real_T *cpu_A, emxArray_real_T *gpu_A,
            boolean_T *A_outdatedOnGpu, const real_T cpu_B_data[],
            emxArray_real_T *cpu_C, boolean_T *C_outdatedOnCpu,
            emxArray_real_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  real_T *gpu_B_data;
  nvtxRangePushA("#fcn#mtimes#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_B_data, sizeof(real_T)), __FILE__, __LINE__);
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_C, i, &yd_emlrtRTEI);
    profileLoopStart("mtimes_loop_0", __LINE__, (cpu_A->size[0] - 1) + 1, "");
    for (i = 0; i < cpu_A->size[0]; i++) {
      cpu_C->data[i] = 0.0;
      *C_outdatedOnGpu = true;
    }
    profileLoopEnd();
  } else {
    real_T alpha1;
    real_T beta1;
    int32_T i;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_C, i, &xd_emlrtRTEI);
    alpha1 = 1.0;
    beta1 = 0.0;
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(hipMemcpy(gpu_B_data, cpu_B_data, sizeof(real_T),
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
    nvtxMarkA("#cublasCheck#" MW_AT_LINE);
    cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            cpu_A->size[0], 1, cpu_A->size[1],
                            (double *)&alpha1, (double *)&gpu_A->data[0],
                            cpu_A->size[0], (double *)&gpu_B_data[0],
                            cpu_A->size[1], (double *)&beta1,
                            (double *)&gpu_C->data[0], cpu_A->size[0]),
                __FILE__, __LINE__);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_B_data), __FILE__, __LINE__);
  nvtxRangePop();
}

} // namespace blas
} // namespace internal
} // namespace coder

// End of code generation (mtimes.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wt.cu
//
// Code generation for function 'wt'
//

// Include files
#include "wt.h"
#include "bsxfun.h"
#include "colon.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "fft.h"
#include "ifft.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo pd_emlrtRTEI{
    188,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo qd_emlrtRTEI{
    205,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo rd_emlrtRTEI{
    209,  // lineNo
    9,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo sd_emlrtRTEI{
    125,                                                          // lineNo
    9,                                                            // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

// Function Declarations
static __global__ void
cwtfilterbank_wt_kernel26(const int32_T i, const int32_T b, emxArray_real_T x);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel26(
    const int32_T i, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T xtmp;
    int32_T b_j1;
    int32_T j2;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

//
//
namespace coder {
void cwtfilterbank_wt(const cwtfilterbank *self, const emxArray_real_T *x,
                      emxArray_creal_T *varargout_1,
                      emxArray_real_T *varargout_2,
                      emxArray_real_T *varargout_3)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T gpu_xposdft;
  emxArray_creal_T *cpu_xposdft;
  emxArray_creal_T *r;
  emxArray_real_T gpu_r1;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xv;
  emxArray_real_T *cpu_r1;
  emxArray_real_T *cpu_x;
  emxArray_real_T *cpu_xv;
  emxArray_real_T *samples;
  real_T M;
  real_T Norig;
  real_T cf;
  real_T d3;
  real_T sigmaPsi;
  int32_T b_i;
  int32_T i;
  int32_T i1;
  int32_T i3;
  int32_T i5;
  int32_T nd2;
  char_T wname_idx_0;
  boolean_T r1_outdatedOnCpu;
  boolean_T r1_outdatedOnGpu;
  boolean_T xv_outdatedOnCpu;
  boolean_T xv_outdatedOnGpu;
  nvtxRangePushA("#fcn#cwtfilterbank_wt#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_r1);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xv);
  r1_outdatedOnCpu = false;
  r1_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xv, 2, &hb_emlrtRTEI, true);
  xv_outdatedOnCpu = false;
  xv_outdatedOnGpu = false;
  i = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = x->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_xv, i, &hb_emlrtRTEI);
  profileLoopStart("cwtfilterbank_wt_loop_0", __LINE__, (x->size[0] - 1) + 1,
                   "");
  for (i = 0; i < x->size[0]; i++) {
    cpu_xv->data[i] = x->data[i];
    xv_outdatedOnGpu = true;
  }
  profileLoopEnd();
  Norig = self->SignalLength;
  if (self->SignalPad > 0.0) {
    boolean_T validLaunchParams;
    M = self->SignalPad;
    if (M < 1.0) {
      b_i = 0;
    } else {
      b_i = static_cast<int32_T>(M);
    }
    M = (static_cast<real_T>(x->size[0]) - self->SignalPad) + 1.0;
    if (M > x->size[0]) {
      i1 = 0;
      i3 = 1;
      i5 = -1;
    } else {
      i1 = x->size[0] - 1;
      i3 = -1;
      i5 = static_cast<int32_T>(M) - 1;
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_x, 2, &ib_emlrtRTEI, true);
    r1_outdatedOnCpu = false;
    i = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = 1;
    cpu_x->size[1] = b_i;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_x, i, &ib_emlrtRTEI);
    profileLoopStart("cwtfilterbank_wt_loop_2", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      cpu_x->data[i] = x->data[i];
      r1_outdatedOnCpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters((b_i >> 1) - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !r1_outdatedOnCpu);
    if (r1_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwtfilterbank_wt_kernel26#" MW_AT_LINE);
      cwtfilterbank_wt_kernel26<<<grid, block>>>(b_i, (b_i >> 1) - 1, gpu_x);
    }
    r1_outdatedOnCpu = true;
    xv_outdatedOnGpu = false;
    i = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    cpu_xv->size[1] =
        ((x->size[0] + cpu_x->size[1]) + div_s32(i5 - i1, i3)) + 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_xv, i, &lb_emlrtRTEI);
    b_i = cpu_x->size[1];
    profileLoopStart("cwtfilterbank_wt_loop_6", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      if (r1_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_x, &gpu_x);
      }
      r1_outdatedOnCpu = false;
      cpu_xv->data[i] = cpu_x->data[i];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    b_i = x->size[0];
    profileLoopStart("cwtfilterbank_wt_loop_9", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      cpu_xv->data[i + cpu_x->size[1]] = x->data[i];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    b_i = div_s32(i5 - i1, i3);
    profileLoopStart("cwtfilterbank_wt_loop_11", __LINE__, b_i + 1, "");
    for (i = 0; i <= b_i; i++) {
      cpu_xv->data[(i + x->size[0]) + cpu_x->size[1]] = x->data[i1 + i3 * i];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_x);
  }
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_xposdft, 2, &qb_emlrtRTEI, true);
  nvtxMarkA("#fft#" MW_AT_LINE);
  fft(cpu_xv, &xv_outdatedOnCpu, &gpu_xv, &xv_outdatedOnGpu, cpu_xposdft,
      &r1_outdatedOnCpu, &gpu_xposdft, &r1_outdatedOnGpu);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xv);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&r, 2, &vb_emlrtRTEI, true);
  if (r1_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_creal_T(cpu_xposdft, &gpu_xposdft);
  }
  nvtxMarkA("#bsxfun#" MW_AT_LINE);
  bsxfun(cpu_xposdft, self->PsiDFT, r);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_xposdft);
  nvtxMarkA("#ifft#" MW_AT_LINE);
  ifft(r, varargout_1);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&r);
  if (self->SignalPad > 0.0) {
    M = self->SignalPad + 1.0;
    d3 = self->SignalPad + Norig;
    if (M > d3) {
      nd2 = 0;
      i1 = 0;
    } else {
      nd2 = static_cast<int32_T>(M) - 1;
      i1 = static_cast<int32_T>(d3);
    }
    i3 = varargout_1->size[0];
    b_i = i1 - nd2;
    profileLoopStart("cwtfilterbank_wt_loop_3", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      profileLoopStart("cwtfilterbank_wt_loop_4", __LINE__, (i3 - 1) + 1, "");
      for (i5 = 0; i5 < i3; i5++) {
        varargout_1->data[i5 + i3 * i] =
            varargout_1->data[i5 + varargout_1->size[0] * (nd2 + i)];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    i = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[1] = i1 - nd2;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(varargout_1, i, &jb_emlrtRTEI);
  }
  i = varargout_2->size[0];
  varargout_2->size[0] = self->WaveletCenterFrequencies->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(varargout_2, i, &pd_emlrtRTEI);
  profileLoopStart("cwtfilterbank_wt_loop_1", __LINE__,
                   (self->WaveletCenterFrequencies->size[0] - 1) + 1, "");
  for (i = 0; i < self->WaveletCenterFrequencies->size[0]; i++) {
    varargout_2->data[i] = self->WaveletCenterFrequencies->data[i];
  }
  profileLoopEnd();
  wname_idx_0 = cv[static_cast<uint8_T>(self->Wavelet[0]) & 127];
  if (wname_idx_0 == 'm') {
    cf = rtNaN;
  } else if (wname_idx_0 == 'a') {
    cf = 6.0;
    sigmaPsi = 1.4142135623730951;
  } else {
    cf = 5.0;
    sigmaPsi = 5.847705;
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&samples, 1, &qd_emlrtRTEI, true);
  i = samples->size[0];
  samples->size[0] = static_cast<int32_T>(Norig);
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(samples, i, &qd_emlrtRTEI);
  profileLoopStart("cwtfilterbank_wt_loop_5", __LINE__,
                   (static_cast<int32_T>(Norig) - 1) + 1, "");
  for (i = 0; i < static_cast<int32_T>(Norig); i++) {
    samples->data[i] = 0.0;
  }
  profileLoopEnd();
  if (std::isinf(Norig)) {
    M = rtNaN;
  } else if (Norig == 0.0) {
    M = 0.0;
  } else {
    M = std::fmod(Norig, 2.0);
  }
  if (M == 1.0) {
    M = std::ceil(Norig / 2.0);
    if (!(M < 1.0)) {
      profileLoopStart("cwtfilterbank_wt_loop_8", __LINE__,
                       static_cast<int32_T>(M - 1.0) + 1, "");
      for (i = 0; i <= static_cast<int32_T>(M - 1.0); i++) {
        samples->data[i] = static_cast<real_T>(i) + 1.0;
      }
      profileLoopEnd();
    }
    if (M + 1.0 > Norig) {
      nd2 = 0;
    } else {
      nd2 = static_cast<int32_T>(static_cast<uint32_T>(M));
    }
    if (!(M - 1.0 < 1.0)) {
      profileLoopStart("cwtfilterbank_wt_loop_10", __LINE__,
                       static_cast<int32_T>(-(1.0 - (M - 1.0))) + 1, "");
      for (i = 0; i <= static_cast<int32_T>(-(1.0 - (M - 1.0))); i++) {
        samples->data[nd2 + i] = (M - 1.0) - static_cast<real_T>(i);
      }
      profileLoopEnd();
    }
  } else {
    M = Norig / 2.0;
    if (!(M < 1.0)) {
      profileLoopStart("cwtfilterbank_wt_loop_7", __LINE__,
                       static_cast<int32_T>(M - 1.0) + 1, "");
      for (i = 0; i <= static_cast<int32_T>(M - 1.0); i++) {
        samples->data[i] = static_cast<real_T>(i) + 1.0;
      }
      profileLoopEnd();
    }
    M = Norig / 2.0 + 1.0;
    if (M > Norig) {
      nd2 = 0;
    } else {
      nd2 = static_cast<int32_T>(M) - 1;
    }
    M = Norig / 2.0;
    if (!(M < 1.0)) {
      if (std::floor(M) == M) {
        profileLoopStart("cwtfilterbank_wt_loop_12", __LINE__,
                         static_cast<int32_T>(-(1.0 - M)) + 1, "");
        for (i = 0; i <= static_cast<int32_T>(-(1.0 - M)); i++) {
          samples->data[nd2 + i] = M - static_cast<real_T>(i);
        }
        profileLoopEnd();
      } else {
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&cpu_r1, 2, &sd_emlrtRTEI, true);
        nvtxMarkA("#eml_float_colon#" MW_AT_LINE);
        eml_float_colon(M, cpu_r1, &r1_outdatedOnCpu, &gpu_r1,
                        &r1_outdatedOnGpu);
        profileLoopStart("cwtfilterbank_wt_loop_13", __LINE__,
                         (cpu_r1->size[1] - 1) + 1, "");
        for (i = 0; i < cpu_r1->size[1]; i++) {
          if (r1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real_T(cpu_r1, &gpu_r1);
          }
          r1_outdatedOnCpu = false;
          samples->data[nd2 + i] = cpu_r1->data[i];
        }
        profileLoopEnd();
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&cpu_r1);
      }
    }
  }
  M = 6.2831853071795862 / cf / sigmaPsi * (1.0 / self->SamplingFrequency);
  i = varargout_3->size[0];
  varargout_3->size[0] = samples->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(varargout_3, i, &rd_emlrtRTEI);
  profileLoopStart("cwtfilterbank_wt_loop_14", __LINE__,
                   (samples->size[0] - 1) + 1, "");
  for (i = 0; i < samples->size[0]; i++) {
    varargout_3->data[i] = 1.0 / (M * samples->data[i]);
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&samples);
  nd2 = self->WaveletCenterFrequencies->size[0];
  M = self->WaveletCenterFrequencies->data[0];
  d3 = self->WaveletCenterFrequencies->data[0];
  profileLoopStart("cwtfilterbank_wt_loop_15", __LINE__, (nd2 - 2) + 1, "");
  for (b_i = 0; b_i <= nd2 - 2; b_i++) {
    cf = self->WaveletCenterFrequencies->data[b_i + 1];
    if (std::isnan(cf)) {
      r1_outdatedOnCpu = false;
    } else if (std::isnan(M)) {
      r1_outdatedOnCpu = true;
    } else {
      r1_outdatedOnCpu = (M < cf);
    }
    if (r1_outdatedOnCpu) {
      M = cf;
    }
    if (std::isnan(cf)) {
      r1_outdatedOnCpu = false;
    } else if (std::isnan(d3)) {
      r1_outdatedOnCpu = true;
    } else {
      r1_outdatedOnCpu = (d3 < cf);
    }
    if (r1_outdatedOnCpu) {
      d3 = cf;
    }
  }
  profileLoopEnd();
  nd2 = varargout_3->size[0] - 1;
  profileLoopStart("cwtfilterbank_wt_loop_16", __LINE__, nd2 + 1, "");
  for (b_i = 0; b_i <= nd2; b_i++) {
    if (varargout_3->data[b_i] > M) {
      varargout_3->data[b_i] = d3;
    }
  }
  profileLoopEnd();
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xv);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_r1);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (wt.cu)

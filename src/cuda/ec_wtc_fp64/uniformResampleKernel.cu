#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// uniformResampleKernel.cu
//
// Code generation for function 'uniformResampleKernel'
//

// Include files
#include "uniformResampleKernel.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "rt_nonfinite.h"
#include "upfirdn.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo yg_emlrtRTEI{
    59,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo ah_emlrtRTEI{
    17,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo bh_emlrtRTEI{
    23,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo ch_emlrtRTEI{
    55,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo dh_emlrtRTEI{
    13,                  // lineNo
    1,                   // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo eh_emlrtRTEI{
    14,                  // lineNo
    15,                  // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo fh_emlrtRTEI{
    16,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace b_signal {
namespace internal {
namespace resample {
void b_uniformResampleAlongFirstDim(
    emxArray_creal_T *cpu_xIn, boolean_T *xIn_outdatedOnCpu,
    emxArray_creal_T *gpu_xIn, boolean_T *xIn_outdatedOnGpu, real_T opts_p,
    real_T opts_q, emxArray_real_T *cpu_opts_filterWithPadding,
    boolean_T *c_opts_filterWithPadding_outdat,
    emxArray_real_T *gpu_opts_filterWithPadding,
    boolean_T *d_opts_filterWithPadding_outdat, real_T opts_filterDelay)
{
  emxArray_creal_T gpu_yOut;
  emxArray_creal_T gpu_yRow;
  emxArray_creal_T *cpu_yOut;
  emxArray_creal_T *cpu_yRow;
  emxArray_creal_T *yRow;
  emxArray_real_T *b_y;
  emxArray_real_T *y;
  boolean_T yRow_outdatedOnCpu;
  boolean_T yRow_outdatedOnGpu;
  nvtxRangePushA("#fcn#b_uniformResampleAlongFirstDim#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_yRow);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_yOut);
  yRow_outdatedOnCpu = false;
  yRow_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_yOut, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&y, 2, &eh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_yRow, 2, &fh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_y, 2, &eb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&yRow, 1, &bh_emlrtRTEI, true);
  if ((!(opts_p == 1.0)) || (!(opts_q == 1.0))) {
    if ((cpu_xIn->size[0] == 1) || (cpu_xIn->size[1] == 1)) {
      real_T b;
      int32_T b_i;
      int32_T i;
      int32_T u1;
      if ((cpu_xIn->size[0] == 0) || (cpu_xIn->size[1] == 0)) {
        u1 = 0;
      } else {
        i = cpu_xIn->size[0];
        u1 = cpu_xIn->size[1];
        if (i >= u1) {
          u1 = i;
        }
      }
      b = std::ceil(static_cast<real_T>(u1) * opts_p / opts_q);
      if ((cpu_xIn->size[0] == 1) && (cpu_xIn->size[1] == 1)) {
        nvtxMarkA("#b_upfirdn#" MW_AT_LINE);
        b_upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu,
                  cpu_opts_filterWithPadding, c_opts_filterWithPadding_outdat,
                  gpu_opts_filterWithPadding, d_opts_filterWithPadding_outdat,
                  opts_p, opts_q, cpu_yRow, &yRow_outdatedOnCpu, &gpu_yRow,
                  &yRow_outdatedOnGpu);
        if ((cpu_yRow->size[0] == 0) || (cpu_yRow->size[1] == 0)) {
          u1 = 0;
        } else {
          i = cpu_yRow->size[0];
          u1 = cpu_yRow->size[1];
          if (i >= u1) {
            u1 = i;
          }
        }
        b_i = cpu_yRow->size[0] * cpu_yRow->size[1];
        cpu_yRow->size[0] = u1;
        cpu_yRow->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_yRow, b_i, &ah_emlrtRTEI);
      } else {
        nvtxMarkA("#b_upfirdn#" MW_AT_LINE);
        b_upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu,
                  cpu_opts_filterWithPadding, c_opts_filterWithPadding_outdat,
                  gpu_opts_filterWithPadding, d_opts_filterWithPadding_outdat,
                  opts_p, opts_q, cpu_yRow, &yRow_outdatedOnCpu, &gpu_yRow,
                  &yRow_outdatedOnGpu);
      }
      if (std::isnan(b)) {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        b_y->data[0] = rtNaN;
      } else if (b < 1.0) {
        b_y->size[0] = 1;
        b_y->size[1] = 0;
      } else {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        profileLoopStart("b_uniformResampleAlongFirstDim_loop_2", __LINE__,
                         static_cast<int32_T>(b - 1.0) + 1, "");
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          b_y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
        profileLoopEnd();
      }
      i = static_cast<int32_T>(b);
      b_i = yRow->size[0];
      yRow->size[0] = static_cast<int32_T>(b);
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(yRow, b_i, &bh_emlrtRTEI);
      profileLoopStart("b_uniformResampleAlongFirstDim_loop_4", __LINE__,
                       (i - 1) + 1, "");
      for (b_i = 0; b_i < i; b_i++) {
        if (yRow_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal_T(cpu_yRow, &gpu_yRow);
        }
        yRow_outdatedOnCpu = false;
        yRow->data[b_i] =
            cpu_yRow
                ->data[static_cast<int32_T>(opts_filterDelay + b_y->data[b_i]) -
                       1];
      }
      profileLoopEnd();
      i = static_cast<int32_T>(b);
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = static_cast<int32_T>(b);
      cpu_xIn->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_xIn, b_i, &ch_emlrtRTEI);
      profileLoopStart("b_uniformResampleAlongFirstDim_loop_5", __LINE__,
                       (i - 1) + 1, "");
      for (u1 = 0; u1 < i; u1++) {
        if (*xIn_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal_T(cpu_xIn, gpu_xIn);
        }
        cpu_xIn->data[u1] = yRow->data[u1];
        *xIn_outdatedOnCpu = false;
        *xIn_outdatedOnGpu = true;
      }
      profileLoopEnd();
    } else {
      real_T b;
      int32_T b_i;
      int32_T i;
      b = std::ceil(static_cast<real_T>(cpu_xIn->size[0]) * opts_p / opts_q);
      nvtxMarkA("#b_upfirdn#" MW_AT_LINE);
      b_upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
                c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
                d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yOut,
                &yRow_outdatedOnCpu, &gpu_yOut, &yRow_outdatedOnGpu);
      if (std::isnan(b)) {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        y->data[0] = rtNaN;
      } else if (b < 1.0) {
        y->size[0] = 1;
        y->size[1] = 0;
      } else {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        profileLoopStart("b_uniformResampleAlongFirstDim_loop_0", __LINE__,
                         static_cast<int32_T>(b - 1.0) + 1, "");
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
        profileLoopEnd();
      }
      i = cpu_yOut->size[1];
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = y->size[1];
      cpu_xIn->size[1] = cpu_yOut->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_xIn, b_i, &yg_emlrtRTEI);
      profileLoopStart("b_uniformResampleAlongFirstDim_loop_1", __LINE__,
                       (i - 1) + 1, "");
      for (b_i = 0; b_i < i; b_i++) {
        profileLoopStart("b_uniformResampleAlongFirstDim_loop_3", __LINE__,
                         (y->size[1] - 1) + 1, "");
        for (int32_T u1{0}; u1 < y->size[1]; u1++) {
          if (*xIn_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_creal_T(cpu_xIn, gpu_xIn);
          }
          if (yRow_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_creal_T(cpu_yOut, &gpu_yOut);
          }
          yRow_outdatedOnCpu = false;
          cpu_xIn->data[u1 + cpu_xIn->size[0] * b_i] =
              cpu_yOut
                  ->data[(static_cast<int32_T>(opts_filterDelay + y->data[u1]) +
                          cpu_yOut->size[0] * b_i) -
                         1];
          *xIn_outdatedOnCpu = false;
          *xIn_outdatedOnGpu = true;
        }
        profileLoopEnd();
      }
      profileLoopEnd();
    }
  }
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&yRow);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_yRow);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_yOut);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_yOut);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_yRow);
  nvtxRangePop();
}

//
//
void uniformResampleAlongFirstDim(
    emxArray_real_T *cpu_xIn, boolean_T *xIn_outdatedOnCpu,
    emxArray_real_T *gpu_xIn, boolean_T *xIn_outdatedOnGpu, real_T opts_p,
    real_T opts_q, emxArray_real_T *cpu_opts_filterWithPadding,
    boolean_T *c_opts_filterWithPadding_outdat,
    emxArray_real_T *gpu_opts_filterWithPadding,
    boolean_T *d_opts_filterWithPadding_outdat, real_T opts_filterDelay)
{
  emxArray_real_T gpu_yOut;
  emxArray_real_T gpu_yRow;
  emxArray_real_T *b_y;
  emxArray_real_T *cpu_yOut;
  emxArray_real_T *cpu_yRow;
  emxArray_real_T *y;
  emxArray_real_T *yRow;
  boolean_T yRow_outdatedOnCpu;
  boolean_T yRow_outdatedOnGpu;
  nvtxRangePushA("#fcn#uniformResampleAlongFirstDim#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_yRow);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_yOut);
  yRow_outdatedOnCpu = false;
  yRow_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_yOut, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&y, 2, &eh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_yRow, 2, &fh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_y, 2, &eb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&yRow, 1, &bh_emlrtRTEI, true);
  if ((!(opts_p == 1.0)) || (!(opts_q == 1.0))) {
    if ((cpu_xIn->size[0] == 1) || (cpu_xIn->size[1] == 1)) {
      real_T b;
      int32_T b_i;
      int32_T i;
      int32_T u1;
      if ((cpu_xIn->size[0] == 0) || (cpu_xIn->size[1] == 0)) {
        u1 = 0;
      } else {
        i = cpu_xIn->size[0];
        u1 = cpu_xIn->size[1];
        if (i >= u1) {
          u1 = i;
        }
      }
      b = std::ceil(static_cast<real_T>(u1) * opts_p / opts_q);
      if ((cpu_xIn->size[0] == 1) && (cpu_xIn->size[1] == 1)) {
        nvtxMarkA("#upfirdn#" MW_AT_LINE);
        upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
                c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
                d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
        if ((cpu_yRow->size[0] == 0) || (cpu_yRow->size[1] == 0)) {
          u1 = 0;
        } else {
          i = cpu_yRow->size[0];
          u1 = cpu_yRow->size[1];
          if (i >= u1) {
            u1 = i;
          }
        }
        b_i = cpu_yRow->size[0] * cpu_yRow->size[1];
        cpu_yRow->size[0] = u1;
        cpu_yRow->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_yRow, b_i, &ah_emlrtRTEI);
      } else {
        nvtxMarkA("#upfirdn#" MW_AT_LINE);
        upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
                c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
                d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
      }
      if (std::isnan(b)) {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        b_y->data[0] = rtNaN;
      } else if (b < 1.0) {
        b_y->size[0] = 1;
        b_y->size[1] = 0;
      } else {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        profileLoopStart("uniformResampleAlongFirstDim_loop_2", __LINE__,
                         static_cast<int32_T>(b - 1.0) + 1, "");
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          b_y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
        profileLoopEnd();
      }
      i = static_cast<int32_T>(b);
      b_i = yRow->size[0];
      yRow->size[0] = static_cast<int32_T>(b);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(yRow, b_i, &bh_emlrtRTEI);
      profileLoopStart("uniformResampleAlongFirstDim_loop_4", __LINE__,
                       (i - 1) + 1, "");
      for (b_i = 0; b_i < i; b_i++) {
        if (yRow_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_yRow, &gpu_yRow);
        }
        yRow_outdatedOnCpu = false;
        yRow->data[b_i] =
            cpu_yRow
                ->data[static_cast<int32_T>(opts_filterDelay + b_y->data[b_i]) -
                       1];
      }
      profileLoopEnd();
      i = static_cast<int32_T>(b);
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = static_cast<int32_T>(b);
      cpu_xIn->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_xIn, b_i, &ch_emlrtRTEI);
      profileLoopStart("uniformResampleAlongFirstDim_loop_5", __LINE__,
                       (i - 1) + 1, "");
      for (u1 = 0; u1 < i; u1++) {
        if (*xIn_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_xIn, gpu_xIn);
        }
        cpu_xIn->data[u1] = yRow->data[u1];
        *xIn_outdatedOnCpu = false;
        *xIn_outdatedOnGpu = true;
      }
      profileLoopEnd();
    } else {
      real_T b;
      int32_T b_i;
      int32_T i;
      b = std::ceil(static_cast<real_T>(cpu_xIn->size[0]) * opts_p / opts_q);
      nvtxMarkA("#upfirdn#" MW_AT_LINE);
      upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
              c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
              d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yOut,
              &yRow_outdatedOnCpu, &gpu_yOut, &yRow_outdatedOnGpu);
      if (std::isnan(b)) {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        y->data[0] = rtNaN;
      } else if (b < 1.0) {
        y->size[0] = 1;
        y->size[1] = 0;
      } else {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        profileLoopStart("uniformResampleAlongFirstDim_loop_0", __LINE__,
                         static_cast<int32_T>(b - 1.0) + 1, "");
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
        profileLoopEnd();
      }
      i = cpu_yOut->size[1];
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = y->size[1];
      cpu_xIn->size[1] = cpu_yOut->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_xIn, b_i, &yg_emlrtRTEI);
      profileLoopStart("uniformResampleAlongFirstDim_loop_1", __LINE__,
                       (i - 1) + 1, "");
      for (b_i = 0; b_i < i; b_i++) {
        profileLoopStart("uniformResampleAlongFirstDim_loop_3", __LINE__,
                         (y->size[1] - 1) + 1, "");
        for (int32_T u1{0}; u1 < y->size[1]; u1++) {
          if (*xIn_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real_T(cpu_xIn, gpu_xIn);
          }
          if (yRow_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real_T(cpu_yOut, &gpu_yOut);
          }
          yRow_outdatedOnCpu = false;
          cpu_xIn->data[u1 + cpu_xIn->size[0] * b_i] =
              cpu_yOut
                  ->data[(static_cast<int32_T>(opts_filterDelay + y->data[u1]) +
                          cpu_yOut->size[0] * b_i) -
                         1];
          *xIn_outdatedOnCpu = false;
          *xIn_outdatedOnGpu = true;
        }
        profileLoopEnd();
      }
      profileLoopEnd();
    }
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&yRow);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_yRow);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_yOut);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_yOut);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_yRow);
  nvtxRangePop();
}

} // namespace resample
} // namespace internal
} // namespace b_signal
} // namespace coder

// End of code generation (uniformResampleKernel.cu)

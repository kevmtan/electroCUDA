#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// circshift.cu
//
// Code generation for function 'circshift'
//

// Include files
#include "circshift.h"
#include "ec_wtc_fp_mexutil.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static __global__ void circshift_kernel203(const int32_T shiftLen,
                                           const int32_T b, int32_T a[2]);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void circshift_kernel203(
    const int32_T shiftLen, const int32_T b, int32_T a[2])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = a[0];
    a[0] = a[1 - shiftLen];
    a[1 - shiftLen] = i2;
  }
}

//
//
namespace coder {
void circshift(int32_T cpu_a[2], boolean_T *a_outdatedOnCpu, int32_T gpu_a[2],
               boolean_T *a_outdatedOnGpu, int32_T p)
{
  dim3 block;
  dim3 grid;
  int32_T shiftLen;
  nvtxRangePushA("#fcn#circshift#" MW_AT_LOCATION);
  shiftLen = p;
  if (p < 0) {
    shiftLen = 1;
  }
  if (shiftLen < 2 - shiftLen) {
    boolean_T validLaunchParams;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(
        computeNumIters(
            static_cast<int32_T>(static_cast<real_T>(2 - shiftLen) / 2.0) - 1),
        &grid, &block, 2147483647U);
    if (*a_outdatedOnGpu) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(hipMemcpy(gpu_a, cpu_a, 8UL, hipMemcpyHostToDevice),
                     __FILE__, __LINE__);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#circshift_kernel203#" MW_AT_LINE);
      circshift_kernel203<<<grid, block>>>(
          shiftLen,
          static_cast<int32_T>(static_cast<real_T>(2 - shiftLen) / 2.0) - 1,
          gpu_a);
    }
    *a_outdatedOnCpu = true;
  }
  if (*a_outdatedOnCpu) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(hipMemcpy(cpu_a, gpu_a, 8UL, hipMemcpyDeviceToHost),
                   __FILE__, __LINE__);
  }
  shiftLen = cpu_a[0];
  cpu_a[0] = cpu_a[1];
  cpu_a[1] = shiftLen;
  *a_outdatedOnCpu = false;
  *a_outdatedOnGpu = true;
  nvtxRangePop();
}

} // namespace coder

// End of code generation (circshift.cu)

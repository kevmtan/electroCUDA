#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wtc_fp.cu
//
// Code generation for function 'ec_wtc_fp'
//

// Include files
#include "ec_wtc_fp.h"
#include "conv2.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "firls.h"
#include "kaiser.h"
#include "mod.h"
#include "rat.h"
#include "resample.h"
#include "rt_nonfinite.h"
#include "uniformResampleKernel.h"
#include "wcoherence.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hip/hip_math_constants.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    1,                                                       // lineNo
    20,                                                      // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    19,                                                      // lineNo
    21,                                                      // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    26,                                                      // lineNo
    29,                                                      // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    26,                                                      // lineNo
    41,                                                      // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    37,                                                      // lineNo
    1,                                                       // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    37,                                                      // lineNo
    9,                                                       // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    26,                                                      // lineNo
    12,                                                      // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    26,                                                      // lineNo
    6,                                                       // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    41,                                                      // lineNo
    5,                                                       // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    42,                                                      // lineNo
    19,                                                      // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    33,                                                      // lineNo
    14,                                                      // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    46,                                                      // lineNo
    6,                                                       // colNo
    "ec_wtc_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wtc_fp.m" // pName
};

// Function Declarations
static __global__ void ec_wtc_fp_kernel1(const int32_T c, const int32_T vlen,
                                         const emxArray_real_T x,
                                         const int32_T b_x, emxArray_real_T x2,
                                         emxArray_real_T x1, int32_T x_dim0);

static __global__ void ec_wtc_fp_kernel10(const emxArray_real_T opts_filter,
                                          const emxArray_int32_T iv1,
                                          const int32_T b_iv1,
                                          emxArray_real_T h);

static __global__ void ec_wtc_fp_kernel11(const emxArray_real_T opts_x,
                                          const int32_T b_opts_x,
                                          emxArray_real_T b_y1);

static __global__ void ec_wtc_fp_kernel12(const emxArray_real_T b_y1,
                                          const int32_T b,
                                          emxArray_real_T b_dv);

static __global__ void ec_wtc_fp_kernel13(const emxArray_creal_T yp,
                                          const int32_T b, emxArray_real_T y);

static __global__ void ec_wtc_fp_kernel14(const emxArray_real_T opts_x,
                                          const int32_T b_opts_x,
                                          emxArray_real_T b_dv);

static __global__ void ec_wtc_fp_kernel15(const emxArray_real_T w,
                                          const int32_T vlen, const int32_T b,
                                          emxArray_real_T y);

static __global__ void ec_wtc_fp_kernel16(emxArray_real_T y);

static __global__ void
ec_wtc_fp_kernel17(const emxArray_real_T y, const emxArray_real_T b_y,
                   const real_T opts_filterDelay, const int32_T ii,
                   const int32_T b, emxArray_real_T b_y1, int32_T y1_dim0);

static __global__ void ec_wtc_fp_kernel18(const int32_T b, emxArray_real_T y);

static __global__ void ec_wtc_fp_kernel19(const emxArray_real_T b_y1,
                                          const int32_T b,
                                          emxArray_real_T b_dv);

static __global__ void ec_wtc_fp_kernel2(const emxArray_real_T xp,
                                         const int32_T b_xp, const int32_T c_xp,
                                         emxArray_real_T b_dv, int32_T dv_dim0,
                                         int32_T xp_dim0);

static __global__ void ec_wtc_fp_kernel20(const emxArray_real_T b_y1,
                                          const int32_T c_y1,
                                          const int32_T d_y1,
                                          emxArray_real_T b_dv, int32_T dv_dim0,
                                          int32_T y1_dim0);

static __global__ void ec_wtc_fp_kernel21(const emxArray_real_T opts_x,
                                          const int32_T b_opts_x,
                                          emxArray_real_T b_y1);

static __global__ void ec_wtc_fp_kernel22(const int32_T b, emxArray_uint32_T y);

static __global__ void ec_wtc_fp_kernel23(const emxArray_real_T b_dv,
                                          const int32_T c_dv,
                                          emxArray_real_T opts_x);

static __global__ void ec_wtc_fp_kernel24(const emxArray_real_T b_dv,
                                          const int32_T c_dv,
                                          const int32_T d_dv,
                                          emxArray_real_T opts_x,
                                          int32_T opts_x_dim0, int32_T dv_dim0);

static __global__ void ec_wtc_fp_kernel3(const emxArray_creal_T yp,
                                         const int32_T b_yp, const int32_T c_yp,
                                         emxArray_creal_T d_yp, int32_T yp_dim0,
                                         int32_T b_yp_dim0);

static __global__ void ec_wtc_fp_kernel4(const emxArray_real_T b_dv,
                                         const int32_T b,
                                         emxArray_real_T opts_x);

static __global__ void ec_wtc_fp_kernel5(const emxArray_real_T r,
                                         const int32_T b_r,
                                         emxArray_real_T b_dv2);

static __global__ void ec_wtc_fp_kernel6(const emxArray_real_T dv3,
                                         const emxArray_real_T b_dv2,
                                         const int32_T c_dv2,
                                         emxArray_real_T h1);

static __global__ void ec_wtc_fp_kernel7(const real_T pqmax,
                                         const emxArray_real_T h1,
                                         const real_T opts_p,
                                         const int32_T b_h1,
                                         emxArray_real_T opts_filter);

static __global__ void ec_wtc_fp_kernel8(const int32_T nZeroBegin,
                                         emxArray_real_T h);

static __global__ void ec_wtc_fp_kernel9(const emxArray_uint32_T y,
                                         const real_T nZeroBegin,
                                         const int32_T b_y,
                                         emxArray_int32_T iv1);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel1(
    const int32_T c, const int32_T vlen, const emxArray_real_T x,
    const int32_T b_x, emxArray_real_T x2, emxArray_real_T x1, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    x1.data[i1] = x.data[i1 + x_dim0 * (vlen - 1)];
    x2.data[i1] = x.data[i1 + x_dim0 * (c - 1)];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel10(
    const emxArray_real_T opts_filter, const emxArray_int32_T iv1,
    const int32_T b_iv1, emxArray_real_T h)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_iv1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h.data[iv1.data[i1] - 1] = opts_filter.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel11(
    const emxArray_real_T opts_x, const int32_T b_opts_x, emxArray_real_T b_y1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_opts_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1] = opts_x.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel12(
    const emxArray_real_T b_y1, const int32_T b, emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_dv.data[i1] = b_y1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel13(
    const emxArray_creal_T yp, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = atan2(yp.data[k].im, yp.data[k].re);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel14(
    const emxArray_real_T opts_x, const int32_T b_opts_x, emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_opts_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_dv.data[i1] = opts_x.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel15(
    const emxArray_real_T w, const int32_T vlen, const int32_T b,
    emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = w.data[vlen * i1];
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_wtc_fp_kernel16(emxArray_real_T y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    y.data[0] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel17(
    const emxArray_real_T y, const emxArray_real_T b_y,
    const real_T opts_filterDelay, const int32_T ii, const int32_T b,
    emxArray_real_T b_y1, int32_T y1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1 + y1_dim0 * ii] =
        y.data[static_cast<int32_T>(opts_filterDelay + b_y.data[i1]) - 1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_wtc_fp_kernel18(const int32_T b,
                                                       emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<real_T>(i1) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel19(
    const emxArray_real_T b_y1, const int32_T b, emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_dv.data[i1] = b_y1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel2(
    const emxArray_real_T xp, const int32_T b_xp, const int32_T c_xp,
    emxArray_real_T b_dv, int32_T dv_dim0, int32_T xp_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_xp) + 1UL) *
                (static_cast<uint64_T>(b_xp) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_xp) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_xp) + 1UL));
    b_dv.data[k + dv_dim0 * i1] = xp.data[i1 + xp_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel20(
    const emxArray_real_T b_y1, const int32_T c_y1, const int32_T d_y1,
    emxArray_real_T b_dv, int32_T dv_dim0, int32_T y1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(d_y1) + 1UL) *
                (static_cast<uint64_T>(c_y1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(c_y1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(c_y1) + 1UL));
    b_dv.data[k + dv_dim0 * i1] = b_y1.data[i1 + y1_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel21(
    const emxArray_real_T opts_x, const int32_T b_opts_x, emxArray_real_T b_y1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_opts_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1] = opts_x.data[i1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_wtc_fp_kernel22(const int32_T b,
                                                       emxArray_uint32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<uint32_T>(i1) + 1U;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel23(
    const emxArray_real_T b_dv, const int32_T c_dv, emxArray_real_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = b_dv.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel24(
    const emxArray_real_T b_dv, const int32_T c_dv, const int32_T d_dv,
    emxArray_real_T opts_x, int32_T opts_x_dim0, int32_T dv_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(d_dv) + 1UL) *
                (static_cast<uint64_T>(c_dv) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(c_dv) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(c_dv) + 1UL));
    opts_x.data[k + opts_x_dim0 * i1] = b_dv.data[i1 + dv_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel3(
    const emxArray_creal_T yp, const int32_T b_yp, const int32_T c_yp,
    emxArray_creal_T d_yp, int32_T yp_dim0, int32_T b_yp_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_yp) + 1UL) *
                (static_cast<uint64_T>(b_yp) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_yp) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_yp) + 1UL));
    d_yp.data[k + yp_dim0 * i1].re = yp.data[i1 + b_yp_dim0 * k].re;
    d_yp.data[k + yp_dim0 * i1].im = -yp.data[i1 + b_yp_dim0 * k].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel4(
    const emxArray_real_T b_dv, const int32_T b, emxArray_real_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = b_dv.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel5(
    const emxArray_real_T r, const int32_T b_r, emxArray_real_T b_dv2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_dv2.data[i1] = r.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel6(
    const emxArray_real_T dv3, const emxArray_real_T b_dv2, const int32_T c_dv2,
    emxArray_real_T h1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_dv2);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h1.data[i1] = b_dv2.data[i1] * dv3.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel7(
    const real_T pqmax, const emxArray_real_T h1, const real_T opts_p,
    const int32_T b_h1, emxArray_real_T opts_filter)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_h1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_filter.data[i1] = opts_p * h1.data[i1] / pqmax;
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_wtc_fp_kernel8(const int32_T nZeroBegin,
                                                      emxArray_real_T h)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nZeroBegin);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h.data[i1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtc_fp_kernel9(
    const emxArray_uint32_T y, const real_T nZeroBegin, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    iv1.data[i1] =
        static_cast<int32_T>(nZeroBegin + static_cast<real_T>(y.data[i1]));
  }
}

//
// function [xx,yy] = ec_wtc_fp(x,c,fs,fLims,fVoices,ds)
void ec_wtc_fp(const emxArray_real_T *cpu_x, const emxArray_uint16_T *c,
               real_T fs, const real_T fLims[2], real_T fVoices, real_T ds[2],
               emxArray_cell_wrap_0 *xx, emxArray_cell_wrap_0 *yy)
{
  static const int32_T iv2[2]{1, 7};
  static const int32_T iv3[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_yp;
  emxArray_creal_T c_gpu_yp;
  emxArray_creal_T gpu_yp;
  emxArray_creal_T *b_cpu_yp;
  emxArray_creal_T *c_cpu_yp;
  emxArray_creal_T *cpu_yp;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T *cpu_iv1;
  emxArray_real_T b_gpu_y;
  emxArray_real_T b_yCol;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_y;
  emxArray_real_T gpu_dv;
  emxArray_real_T gpu_dv2;
  emxArray_real_T gpu_dv3;
  emxArray_real_T gpu_h;
  emxArray_real_T gpu_h1;
  emxArray_real_T gpu_opts_filter;
  emxArray_real_T gpu_opts_x;
  emxArray_real_T gpu_r;
  emxArray_real_T gpu_w;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_x1;
  emxArray_real_T gpu_x2;
  emxArray_real_T gpu_xp;
  emxArray_real_T gpu_y1;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_dv;
  emxArray_real_T *cpu_dv2;
  emxArray_real_T *cpu_dv3;
  emxArray_real_T *cpu_h;
  emxArray_real_T *cpu_h1;
  emxArray_real_T *cpu_opts_filter;
  emxArray_real_T *cpu_opts_x;
  emxArray_real_T *cpu_r;
  emxArray_real_T *cpu_w;
  emxArray_real_T *cpu_x1;
  emxArray_real_T *cpu_x2;
  emxArray_real_T *cpu_xp;
  emxArray_real_T *cpu_y;
  emxArray_real_T *cpu_y1;
  emxArray_real_T *d_cpu_y;
  emxArray_real_T *yCol;
  emxArray_uint32_T gpu_y;
  emxArray_uint32_T *b_cpu_y;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T Ly;
  real_T opts_q;
  int32_T opts_x[1];
  int32_T b_i1;
  int32_T i;
  int32_T i1;
  int32_T i3;
  int32_T i4;
  char_T unusedExpr[23];
  boolean_T b_y_needsGpuEnsureCapacity;
  boolean_T b_yp_outdatedOnCpu;
  boolean_T dv_outdatedOnCpu;
  boolean_T h_outdatedOnCpu;
  boolean_T h_outdatedOnGpu;
  boolean_T opts_isRowVectorInput;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  boolean_T yp_needsGpuEnsureCapacity;
  boolean_T yp_outdatedOnCpu;
  boolean_T yp_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_wtc_fp#" MW_AT_LOCATION "#M101,101,100");
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&c_gpu_yp);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_w);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_y1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_uint32_T#" MW_AT_LINE);
  gpuEmxReset_uint32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_h);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_h1);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv3);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_opts_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_opts_filter);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_yp);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_yp);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xp);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x1);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x);
  w_outdatedOnCpu = false;
  w_outdatedOnGpu = false;
  yp_outdatedOnCpu = false;
  yp_outdatedOnGpu = false;
  x_outdatedOnGpu = true;
  b_y_needsGpuEnsureCapacity = true;
  y_needsGpuEnsureCapacity = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  electroCUDA: run wavelet coherence (DEPRECIATED SOON)
  // 'ec_wtc_fp:4' x (:,:){mustBeFloat}
  // 'ec_wtc_fp:5' c (:,2) uint16
  // 'ec_wtc_fp:6' fs (1,1) double
  // 'ec_wtc_fp:7' fLims (1,2) double
  // 'ec_wtc_fp:8' fVoices (1,1) double = 12
  // 'ec_wtc_fp:9' ds (1,2) double = [0 0]
  //  Trigger kernel creation
  // 'ec_wtc_fp:13' coder.gpu.kernelfun;
  //  Make vars
  // 'ec_wtc_fp:16' nPairs = height(c);
  // 'ec_wtc_fp:17' tmp = cell(1,nPairs);
  i = xx->size[0] * xx->size[1];
  xx->size[0] = 1;
  xx->size[1] = c->size[0];
  nvtxMarkA("#emxEnsureCapacity_cell_wrap_0#" MW_AT_LINE);
  emxEnsureCapacity_cell_wrap_0(xx, i, &emlrtRTEI);
  // 'ec_wtc_fp:18' xx = coder.nullcopy(tmp);
  //  Fix define array error
  // 'ec_wtc_fp:19' yy = coder.nullcopy(tmp);
  i = yy->size[0] * yy->size[1];
  yy->size[0] = 1;
  yy->size[1] = c->size[0];
  nvtxMarkA("#emxEnsureCapacity_cell_wrap_0#" MW_AT_LINE);
  emxEnsureCapacity_cell_wrap_0(yy, i, &b_emlrtRTEI);
  // 'ec_wtc_fp:20' if ds(1)<=0 || ds(2)<=1 || ~(ds(2)>ds(1))
  if ((ds[0] <= 0.0) || (ds[1] <= 1.0) || (!(ds[1] > ds[0]))) {
    // 'ec_wtc_fp:21' ds = [0 0];
    profileLoopStart("ec_wtc_fp_loop_0", __LINE__, 1 + 1, "");
    for (i1 = 0; i1 < 2; i1++) {
      ds[i1] = 0.0;
    }
    profileLoopEnd();
  }
  //  Loop across pairs
  // 'ec_wtc_fp:25' for p = 1:nPairs
  b_i1 = c->size[0];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_x1, 1, &c_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_x2, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xp, 2, &emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_yp, 2, &emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv, 2, &emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_yp, 2, &x_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_y, 2, &y_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_opts_filter, 1, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_opts_x, 2, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv2, 1, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv3, 1, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_h1, 1, &l_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_h, 1, &m_emlrtRTEI, true);
  nvtxMarkA("#emxInit_uint32_T#" MW_AT_LINE);
  emxInit_uint32_T(&b_cpu_y, 2, &ab_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 2, &bb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_y1, 2, &cb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&yCol, 1, &db_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_w, 1, &v_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_y, 1, &v_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_y, 2, &eb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_r, 2, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&c_cpu_yp, 2, &u_emlrtRTEI, true);
  profileLoopStart("ec_wtc_fp_loop_1", __LINE__, (b_i1 - 1) + 1, "");
  for (int32_T p{0}; p < b_i1; p++) {
    int32_T b_c;
    int32_T vlen;
    uint32_T b_dv1[2];
    boolean_T validLaunchParams;
    // 'ec_wtc_fp:26' [xx{p},yy{p}] =
    // wtc_lfn(x(:,c(p,1)),x(:,c(p,2)),fs,fLims,fVoices,ds);
    vlen = c->data[p];
    i = cpu_x1->size[0];
    cpu_x1->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_x1, i, &c_emlrtRTEI);
    b_c = c->data[p + c->size[0]];
    i = cpu_x2->size[0];
    cpu_x2->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_x2, i, &d_emlrtRTEI);
    i = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (x_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x2, &gpu_x2, true);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x1, &gpu_x1, true);
    if (x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_wtc_fp_kernel1#" MW_AT_LINE);
      ec_wtc_fp_kernel1<<<grid, block>>>(b_c, vlen, gpu_x, i, gpu_x2, gpu_x1,
                                         cpu_x->size[0U]);
    }
    //  Wavelet coherence - local function
    //  Calculate wavelet coherence
    // 'ec_wtc_fp:36' [xp,yp] =
    // wcoherence(x1,x2,fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_x1, &gpu_x1);
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_x2, &gpu_x2);
    nvtxMarkA("#wcoherence#" MW_AT_LINE);
    coder::wcoherence(cpu_x1, cpu_x2, fs, fLims, fVoices, cpu_xp, cpu_yp,
                      &yp_outdatedOnCpu, &gpu_yp, &yp_outdatedOnGpu);
    // 'ec_wtc_fp:37' xp=xp';
    i = cpu_dv->size[0] * cpu_dv->size[1];
    cpu_dv->size[0] = cpu_xp->size[1];
    cpu_dv->size[1] = cpu_xp->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_dv, i, &e_emlrtRTEI);
    i = cpu_xp->size[0] - 1;
    i1 = cpu_xp->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i, i1), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_xp, &gpu_xp, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_xp, cpu_xp);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_wtc_fp_kernel2#" MW_AT_LINE);
      ec_wtc_fp_kernel2<<<grid, block>>>(gpu_xp, i1, i, gpu_dv,
                                         cpu_dv->size[0U], cpu_xp->size[0U]);
    }
    dv_outdatedOnCpu = true;
    // 'ec_wtc_fp:37' yp=yp';
    i = b_cpu_yp->size[0] * b_cpu_yp->size[1];
    b_cpu_yp->size[0] = cpu_yp->size[1];
    b_cpu_yp->size[1] = cpu_yp->size[0];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(b_cpu_yp, i, &f_emlrtRTEI);
    i = cpu_yp->size[0] - 1;
    i1 = cpu_yp->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i, i1), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_yp, &gpu_yp, !yp_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(b_cpu_yp, &b_gpu_yp, true);
    yp_needsGpuEnsureCapacity = false;
    if (yp_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_yp, cpu_yp);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_wtc_fp_kernel3#" MW_AT_LINE);
      ec_wtc_fp_kernel3<<<grid, block>>>(gpu_yp, i1, i, b_gpu_yp,
                                         b_cpu_yp->size[0U], cpu_yp->size[0U]);
    }
    yp_outdatedOnGpu = false;
    b_yp_outdatedOnCpu = true;
    //  make column-major
    //  Downsample
    // 'ec_wtc_fp:40' if ds(2)
    if (ds[1] != 0.0) {
      real_T b_dv[4];
      real_T fc;
      real_T nZeroBegin;
      real_T opts_filterDelay;
      real_T opts_p;
      real_T pqmax;
      int32_T opts_dim;
      boolean_T opts_x_outdatedOnCpu;
      // 'ec_wtc_fp:41' xp = resample(xp,ds(1),ds(2));
      if ((cpu_dv->size[0] == 1) || (cpu_dv->size[1] == 1)) {
        opts_dim = 1;
      } else {
        opts_dim = 2;
        if (cpu_dv->size[0] != 1) {
          opts_dim = 1;
        }
      }
      opts_isRowVectorInput = (cpu_dv->size[0] == 1);
      if (opts_dim == 1) {
        if ((cpu_dv->size[0] == 1) || (cpu_dv->size[1] == 1)) {
          if ((cpu_dv->size[0] == 0) || (cpu_dv->size[1] == 0)) {
            b_c = 0;
          } else {
            vlen = cpu_dv->size[0];
            b_c = cpu_dv->size[1];
            if (vlen >= b_c) {
              b_c = vlen;
            }
          }
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
          cpu_opts_x->size[0] = b_c;
          cpu_opts_x->size[1] = 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_opts_x, i, &g_emlrtRTEI);
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(b_c - 1), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_opts_x, &gpu_opts_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel4#" MW_AT_LINE);
            ec_wtc_fp_kernel4<<<grid, block>>>(gpu_dv, b_c - 1, gpu_opts_x);
          }
          opts_x_outdatedOnCpu = true;
        } else {
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
          cpu_opts_x->size[0] = cpu_dv->size[0];
          cpu_opts_x->size[1] = cpu_dv->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_opts_x, i, &g_emlrtRTEI);
          i1 = cpu_dv->size[0] * cpu_dv->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(i1), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_opts_x, &gpu_opts_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel23#" MW_AT_LINE);
            ec_wtc_fp_kernel23<<<grid, block>>>(gpu_dv, i1, gpu_opts_x);
          }
          opts_x_outdatedOnCpu = true;
        }
      } else {
        i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
        cpu_opts_x->size[0] = cpu_dv->size[1];
        cpu_opts_x->size[1] = cpu_dv->size[0];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_opts_x, i, &g_emlrtRTEI);
        i1 = cpu_dv->size[0] - 1;
        i = cpu_dv->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i1, i), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_opts_x, &gpu_opts_x, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wtc_fp_kernel24#" MW_AT_LINE);
          ec_wtc_fp_kernel24<<<grid, block>>>(gpu_dv, i, i1, gpu_opts_x,
                                              cpu_opts_x->size[0U],
                                              cpu_dv->size[0U]);
        }
        opts_x_outdatedOnCpu = true;
      }
      nvtxMarkA("#rat#" MW_AT_LINE);
      opts_p = coder::rat(ds[0] / ds[1], &opts_q);
      pqmax = std::fmax(opts_p, opts_q);
      fc = 0.5 / pqmax;
      pqmax = 20.0 * pqmax + 1.0;
      b_dv[0] = 0.0;
      b_dv[1] = 2.0 * fc;
      b_dv[2] = 2.0 * fc;
      b_dv[3] = 1.0;
      nvtxMarkA("#firls#" MW_AT_LINE);
      coder::firls(pqmax - 1.0, b_dv, cpu_r);
      i = cpu_dv2->size[0];
      cpu_dv2->size[0] = cpu_r->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_dv2, i, &k_emlrtRTEI);
      i = cpu_r->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_r, &gpu_r, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_dv2, &gpu_dv2, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, cpu_r);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtc_fp_kernel5#" MW_AT_LINE);
        ec_wtc_fp_kernel5<<<grid, block>>>(gpu_r, i, gpu_dv2);
      }
      nvtxMarkA("#kaiser#" MW_AT_LINE);
      coder::kaiser(pqmax, cpu_dv3);
      i = cpu_h1->size[0];
      cpu_h1->size[0] = cpu_dv2->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_h1, i, &l_emlrtRTEI);
      i = cpu_dv2->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_dv3, &gpu_dv3, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_h1, &gpu_h1, true);
      yp_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_dv3, cpu_dv3);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtc_fp_kernel6#" MW_AT_LINE);
        ec_wtc_fp_kernel6<<<grid, block>>>(gpu_dv3, gpu_dv2, i, gpu_h1);
      }
      vlen = cpu_h1->size[0];
      if (cpu_h1->size[0] == 0) {
        pqmax = 0.0;
      } else {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_h1, &gpu_h1);
        pqmax = cpu_h1->data[0];
        yp_needsGpuEnsureCapacity = true;
        profileLoopStart("ec_wtc_fp_loop_5", __LINE__, (vlen - 2) + 1, "");
        for (i1 = 0; i1 <= vlen - 2; i1++) {
          pqmax += cpu_h1->data[i1 + 1];
        }
        profileLoopEnd();
      }
      i = cpu_opts_filter->size[0];
      cpu_opts_filter->size[0] = cpu_h1->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_opts_filter, i, &g_emlrtRTEI);
      i = cpu_h1->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (yp_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_h1, &gpu_h1, true);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_opts_filter, &gpu_opts_filter, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtc_fp_kernel7#" MW_AT_LINE);
        ec_wtc_fp_kernel7<<<grid, block>>>(pqmax, gpu_h1, opts_p, i,
                                           gpu_opts_filter);
      }
      pqmax = (static_cast<real_T>(cpu_opts_filter->size[0]) - 1.0) / 2.0;
      nvtxMarkA("#b_mod#" MW_AT_LINE);
      nZeroBegin = std::floor(opts_q - coder::b_mod(pqmax, opts_q));
      opts_filterDelay = std::floor(std::ceil(pqmax + nZeroBegin) / opts_q);
      pqmax = static_cast<real_T>(cpu_opts_filter->size[0]) + nZeroBegin;
      fc = 0.0;
      vlen = cpu_opts_x->size[0];
      nvtxRangePushA("#loop#ec_wtc_fp_whileloop_0##" MW_AT_LINE);
      while (std::ceil(
                 (((static_cast<real_T>(vlen) - 1.0) * opts_p + pqmax) + fc) /
                 opts_q) -
                 opts_filterDelay <
             std::ceil(static_cast<real_T>(vlen) * opts_p / opts_q)) {
        fc++;
      }
      nvtxRangePop();
      i = cpu_h->size[0];
      cpu_h->size[0] = static_cast<int32_T>(
          (nZeroBegin + static_cast<real_T>(cpu_opts_filter->size[0])) + fc);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_h, i, &m_emlrtRTEI);
      i = static_cast<int32_T>(
              (nZeroBegin + static_cast<real_T>(cpu_opts_filter->size[0])) +
              fc) -
          1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_h, &gpu_h, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtc_fp_kernel8#" MW_AT_LINE);
        ec_wtc_fp_kernel8<<<grid, block>>>(i, gpu_h);
      }
      if (cpu_opts_filter->size[0] < 1) {
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = 0;
      } else {
        uint32_T u;
        u = static_cast<uint32_T>(cpu_opts_filter->size[0]);
        i = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_opts_filter->size[0];
        nvtxMarkA("#emxEnsureCapacity_uint32_T#" MW_AT_LINE);
        emxEnsureCapacity_uint32_T(b_cpu_y, i, &n_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(u) - 1),
                                &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_uint32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_uint32_T(b_cpu_y, &gpu_y, true);
        y_needsGpuEnsureCapacity = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wtc_fp_kernel22#" MW_AT_LINE);
          ec_wtc_fp_kernel22<<<grid, block>>>(static_cast<int32_T>(u) - 1,
                                              gpu_y);
        }
      }
      i = cpu_iv1->size[0] * cpu_iv1->size[1];
      cpu_iv1->size[0] = 1;
      cpu_iv1->size[1] = b_cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_iv1, i, &o_emlrtRTEI);
      i = b_cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (y_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_uint32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_uint32_T(b_cpu_y, &gpu_y, true);
      }
      y_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtc_fp_kernel9#" MW_AT_LINE);
        ec_wtc_fp_kernel9<<<grid, block>>>(gpu_y, nZeroBegin, i, gpu_iv1);
      }
      i = cpu_iv1->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wtc_fp_kernel10#" MW_AT_LINE);
        ec_wtc_fp_kernel10<<<grid, block>>>(gpu_opts_filter, gpu_iv1, i, gpu_h);
      }
      h_outdatedOnGpu = false;
      h_outdatedOnCpu = true;
      if (opts_dim == 1) {
        if (opts_isRowVectorInput) {
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = cpu_opts_x->size[0];
          cpu_y1->size[1] = cpu_opts_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_y1, i, &r_emlrtRTEI);
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_y1, &gpu_y1, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel11#" MW_AT_LINE);
            ec_wtc_fp_kernel11<<<grid, block>>>(gpu_opts_x, i, gpu_y1);
          }
          opts_isRowVectorInput = false;
          yp_needsGpuEnsureCapacity = true;
          nvtxMarkA("#uniformResampleAlongFirstDim#" MW_AT_LINE);
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_y1, &yp_needsGpuEnsureCapacity, &gpu_y1,
              &opts_isRowVectorInput, opts_p, opts_q, cpu_h, &h_outdatedOnCpu,
              &gpu_h, &h_outdatedOnGpu, opts_filterDelay);
          if ((cpu_y1->size[0] == 0) || (cpu_y1->size[1] == 0)) {
            b_c = 0;
          } else {
            vlen = cpu_y1->size[0];
            b_c = cpu_y1->size[1];
            if (vlen >= b_c) {
              b_c = vlen;
            }
          }
          i = cpu_dv->size[0] * cpu_dv->size[1];
          cpu_dv->size[0] = 1;
          cpu_dv->size[1] = b_c;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_dv, i, &q_emlrtRTEI);
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(b_c - 1), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_y1, &gpu_y1, !opts_isRowVectorInput);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          if (opts_isRowVectorInput) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_y1, cpu_y1);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel12#" MW_AT_LINE);
            ec_wtc_fp_kernel12<<<grid, block>>>(gpu_y1, b_c - 1, gpu_dv);
          }
        } else {
          i = cpu_dv->size[0] * cpu_dv->size[1];
          cpu_dv->size[0] = cpu_opts_x->size[0];
          cpu_dv->size[1] = cpu_opts_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_dv, i, &q_emlrtRTEI);
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel14#" MW_AT_LINE);
            ec_wtc_fp_kernel14<<<grid, block>>>(gpu_opts_x, i, gpu_dv);
          }
          yp_needsGpuEnsureCapacity = false;
          nvtxMarkA("#uniformResampleAlongFirstDim#" MW_AT_LINE);
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_dv, &dv_outdatedOnCpu, &gpu_dv, &yp_needsGpuEnsureCapacity,
              opts_p, opts_q, cpu_h, &h_outdatedOnCpu, &gpu_h, &h_outdatedOnGpu,
              opts_filterDelay);
        }
      } else {
        if (cpu_opts_x->size[0] == 1) {
          pqmax = std::ceil(opts_p / opts_q);
          opts_isRowVectorInput = false;
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = static_cast<int32_T>(pqmax);
          cpu_y1->size[1] = cpu_opts_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_y1, i, &s_emlrtRTEI);
          yp_needsGpuEnsureCapacity = true;
          opts_dim = cpu_opts_x->size[1];
          if (cpu_opts_x->size[1] - 1 >= 0) {
            if ((static_cast<int32_T>(opts_p) == 0) ||
                ((static_cast<int32_T>(opts_p) > 0) &&
                 (static_cast<int32_T>(opts_p) < 1))) {
              i3 = 1;
              i4 = 0;
            } else {
              i3 = static_cast<int32_T>(opts_p);
              i4 = static_cast<int32_T>(opts_p);
            }
            Ly = std::ceil(
                (0.0 * opts_p + static_cast<real_T>(cpu_h->size[0])) / opts_q);
          }
          profileLoopStart("ec_wtc_fp_loop_6", __LINE__, (opts_dim - 1) + 1,
                           "");
          for (int32_T ii{0}; ii < opts_dim; ii++) {
            if (!(opts_p - 1.0 >= 0.0)) {
              y = nullptr;
              m = emlrtCreateCharArray(2, &iv2[0]);
              emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
              emlrtAssign(&y, m);
              b_y = nullptr;
              m1 = emlrtCreateDoubleScalar(opts_p - 1.0);
              emlrtAssign(&b_y, m1);
              nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
              nvtxMarkA("#b_sprintf#" MW_AT_LINE);
              emlrt_marshallIn(b_sprintf(y, b_y, &emlrtMCI),
                               "<output of sprintf>", unusedExpr);
            }
            i = yCol->size[0];
            yCol->size[0] = static_cast<int32_T>(opts_p);
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(yCol, i, &t_emlrtRTEI);
            profileLoopStart("ec_wtc_fp_loop_7", __LINE__,
                             (static_cast<int32_T>(opts_p) - 1) + 1, "");
            for (i1 = 0; i1 < static_cast<int32_T>(opts_p); i1++) {
              yCol->data[i1] = 0.0;
            }
            profileLoopEnd();
            nvtxMarkA("#div_s32#" MW_AT_LINE);
            vlen = div_s32(i4 - 1, i3) + 1;
            profileLoopStart("ec_wtc_fp_loop_8", __LINE__, (vlen - 1) + 1, "");
            for (i1 = 0; i1 < vlen; i1++) {
              if (opts_x_outdatedOnCpu) {
                nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
                gpuEmxMemcpyGpuToCpu_real_T(cpu_opts_x, &gpu_opts_x);
              }
              opts_x_outdatedOnCpu = false;
              yCol->data[i3 * i1] = cpu_opts_x->data[ii];
            }
            profileLoopEnd();
            b_yCol = *yCol;
            opts_x[0] = static_cast<int32_T>(opts_p);
            b_yCol.size = &opts_x[0];
            b_yCol.numDimensions = 1;
            nvtxMarkA("#b_conv2#" MW_AT_LINE);
            coder::b_conv2(&b_yCol, cpu_h, &h_outdatedOnCpu, &gpu_h,
                           &h_outdatedOnGpu, cpu_w, &w_outdatedOnCpu, &gpu_w,
                           &w_outdatedOnGpu);
            if (!(opts_q - 1.0 >= 0.0)) {
              c_y = nullptr;
              m2 = emlrtCreateCharArray(2, &iv3[0]);
              emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
              emlrtAssign(&c_y, m2);
              d_y = nullptr;
              m3 = emlrtCreateDoubleScalar(opts_q - 1.0);
              emlrtAssign(&d_y, m3);
              nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
              nvtxMarkA("#b_sprintf#" MW_AT_LINE);
              emlrt_marshallIn(b_sprintf(c_y, d_y, &emlrtMCI),
                               "<output of sprintf>", unusedExpr);
            }
            if (opts_q == 0.0) {
              vlen = 1;
              b_c = -1;
            } else {
              vlen = static_cast<int32_T>(opts_q);
              b_c = cpu_w->size[0] - 1;
            }
            i = c_cpu_y->size[0];
            nvtxMarkA("#div_s32#" MW_AT_LINE);
            c_cpu_y->size[0] = div_s32(b_c, vlen) + 1;
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(c_cpu_y, i, &v_emlrtRTEI);
            i1 = b_c / vlen;
            nvtxMarkA("#computeNumIters#" MW_AT_LINE);
            mwGetLaunchParameters1D(computeNumIters(i1), &grid, &block,
                                    2147483647U);
            nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
            nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
            yp_outdatedOnGpu = false;
            if (w_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyCpuToGpu_real_T(&gpu_w, cpu_w);
            }
            w_outdatedOnGpu = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              nvtxMarkA("#ec_wtc_fp_kernel15#" MW_AT_LINE);
              ec_wtc_fp_kernel15<<<grid, block>>>(gpu_w, vlen, i1, c_gpu_y);
            }
            nvtxMarkA("#div_s32#" MW_AT_LINE);
            if (!(div_s32(b_c, vlen) + 1 < Ly)) {
              i = c_cpu_y->size[0];
              if (Ly < 1.0) {
                c_cpu_y->size[0] = 0;
              } else {
                c_cpu_y->size[0] = static_cast<int32_T>(Ly);
              }
              nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
              emxEnsureCapacity_real_T(c_cpu_y, i, &w_emlrtRTEI);
              yp_outdatedOnGpu = true;
            }
            if (std::isnan(pqmax)) {
              i = d_cpu_y->size[0] * d_cpu_y->size[1];
              d_cpu_y->size[0] = 1;
              d_cpu_y->size[1] = 1;
              nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
              emxEnsureCapacity_real_T(d_cpu_y, i, &n_emlrtRTEI);
              nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
              gpuEmxEnsureCapacity_real_T(d_cpu_y, &d_gpu_y, true);
              b_y_needsGpuEnsureCapacity = false;
              nvtxMarkA("#ec_wtc_fp_kernel16#" MW_AT_LINE);
              ec_wtc_fp_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                  d_gpu_y);
            } else if (pqmax < 1.0) {
              d_cpu_y->size[0] = 1;
              d_cpu_y->size[1] = 0;
            } else {
              i = d_cpu_y->size[0] * d_cpu_y->size[1];
              d_cpu_y->size[0] = 1;
              d_cpu_y->size[1] = static_cast<int32_T>(pqmax - 1.0) + 1;
              nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
              emxEnsureCapacity_real_T(d_cpu_y, i, &n_emlrtRTEI);
              nvtxMarkA("#computeNumIters#" MW_AT_LINE);
              mwGetLaunchParameters1D(
                  computeNumIters(static_cast<int32_T>(pqmax - 1.0)), &grid,
                  &block, 2147483647U);
              nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
              gpuEmxEnsureCapacity_real_T(d_cpu_y, &d_gpu_y, true);
              b_y_needsGpuEnsureCapacity = false;
              validLaunchParams = mwValidateLaunchParameters(grid, block);
              if (validLaunchParams) {
                nvtxMarkA("#ec_wtc_fp_kernel18#" MW_AT_LINE);
                ec_wtc_fp_kernel18<<<grid, block>>>(
                    static_cast<int32_T>(pqmax - 1.0), d_gpu_y);
              }
            }
            vlen = cpu_y1->size[0];
            nvtxMarkA("#computeNumIters#" MW_AT_LINE);
            mwGetLaunchParameters1D(computeNumIters(vlen - 1), &grid, &block,
                                    2147483647U);
            if (yp_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
              gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
            }
            if (b_y_needsGpuEnsureCapacity) {
              nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
              gpuEmxEnsureCapacity_real_T(d_cpu_y, &d_gpu_y, true);
            }
            b_y_needsGpuEnsureCapacity = false;
            if (yp_needsGpuEnsureCapacity) {
              nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
              gpuEmxEnsureCapacity_real_T(cpu_y1, &gpu_y1, true);
            }
            yp_needsGpuEnsureCapacity = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              nvtxMarkA("#ec_wtc_fp_kernel17#" MW_AT_LINE);
              ec_wtc_fp_kernel17<<<grid, block>>>(
                  c_gpu_y, d_gpu_y, opts_filterDelay, ii, vlen - 1, gpu_y1,
                  cpu_y1->size[0U]);
            }
          }
          profileLoopEnd();
        } else {
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = cpu_opts_x->size[0];
          cpu_y1->size[1] = cpu_opts_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_y1, i, &p_emlrtRTEI);
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_y1, &gpu_y1, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel21#" MW_AT_LINE);
            ec_wtc_fp_kernel21<<<grid, block>>>(gpu_opts_x, i, gpu_y1);
          }
          opts_isRowVectorInput = false;
          yp_needsGpuEnsureCapacity = true;
          nvtxMarkA("#uniformResampleAlongFirstDim#" MW_AT_LINE);
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_y1, &yp_needsGpuEnsureCapacity, &gpu_y1,
              &opts_isRowVectorInput, opts_p, opts_q, cpu_h, &h_outdatedOnCpu,
              &gpu_h, &h_outdatedOnGpu, opts_filterDelay);
          yp_needsGpuEnsureCapacity = true;
        }
        if ((cpu_opts_x->size[0] == 1) || (cpu_opts_x->size[1] == 1)) {
          vlen = cpu_dv->size[0];
          b_c = cpu_dv->size[0];
          opts_dim = cpu_y1->size[0];
          i = cpu_dv->size[0] * cpu_dv->size[1];
          cpu_dv->size[0] = vlen;
          cpu_dv->size[1] = cpu_y1->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_dv, i, &q_emlrtRTEI);
          i1 = b_c * opts_dim - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(i1), &grid, &block,
                                  2147483647U);
          if (yp_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real_T(cpu_y1, &gpu_y1,
                                        !opts_isRowVectorInput);
          }
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          if (opts_isRowVectorInput) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_y1, cpu_y1);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel19#" MW_AT_LINE);
            ec_wtc_fp_kernel19<<<grid, block>>>(gpu_y1, i1, gpu_dv);
          }
        } else {
          i = cpu_dv->size[0] * cpu_dv->size[1];
          cpu_dv->size[0] = cpu_y1->size[1];
          cpu_dv->size[1] = cpu_y1->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_dv, i, &q_emlrtRTEI);
          i = cpu_y1->size[0] - 1;
          i1 = cpu_y1->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(i, i1), &grid, &block,
                                  2147483647U);
          if (yp_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real_T(cpu_y1, &gpu_y1,
                                        !opts_isRowVectorInput);
          }
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          if (opts_isRowVectorInput) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_y1, cpu_y1);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wtc_fp_kernel20#" MW_AT_LINE);
            ec_wtc_fp_kernel20<<<grid, block>>>(
                gpu_y1, i1, i, gpu_dv, cpu_dv->size[0U], cpu_y1->size[0U]);
          }
        }
      }
      // 'ec_wtc_fp:42' yp = resample(yp,ds(1),ds(2));
      opts_isRowVectorInput = false;
      i = c_cpu_yp->size[0] * c_cpu_yp->size[1];
      c_cpu_yp->size[0] = b_cpu_yp->size[0];
      c_cpu_yp->size[1] = b_cpu_yp->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(c_cpu_yp, i, &u_emlrtRTEI);
      vlen = b_cpu_yp->size[0] * b_cpu_yp->size[1] - 1;
      profileLoopStart("ec_wtc_fp_loop_9", __LINE__, vlen + 1, "");
      for (i = 0; i <= vlen; i++) {
        if (b_yp_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal_T(b_cpu_yp, &b_gpu_yp);
        }
        b_yp_outdatedOnCpu = false;
        c_cpu_yp->data[i] = b_cpu_yp->data[i];
        opts_isRowVectorInput = true;
      }
      profileLoopEnd();
      nvtxMarkA("#resample#" MW_AT_LINE);
      coder::resample(c_cpu_yp, &c_gpu_yp, &opts_isRowVectorInput, ds[0], ds[1],
                      b_cpu_yp, &b_yp_outdatedOnCpu, &b_gpu_yp,
                      &yp_outdatedOnGpu);
      yp_needsGpuEnsureCapacity = true;
    }
    //  Convert wavelet to phase angle
    // 'ec_wtc_fp:46' yp = angle(yp);
    vlen = b_cpu_yp->size[0] * b_cpu_yp->size[1];
    profileLoopStart("ec_wtc_fp_loop_2", __LINE__, 1 + 1, "");
    for (i1 = 0; i1 < 2; i1++) {
      b_dv1[i1] = static_cast<uint32_T>(b_cpu_yp->size[i1]);
    }
    profileLoopEnd();
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = static_cast<int32_T>(b_dv1[0]);
    cpu_y->size[1] = static_cast<int32_T>(b_dv1[1]);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, i, &h_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(vlen - 1), &grid, &block,
                            2147483647U);
    if (yp_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(b_cpu_yp, &b_gpu_yp, !yp_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_y, &b_gpu_y, true);
    if (yp_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&b_gpu_yp, b_cpu_yp);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_wtc_fp_kernel13#" MW_AT_LINE);
      ec_wtc_fp_kernel13<<<grid, block>>>(b_gpu_yp, vlen - 1, b_gpu_y);
    }
    yp_needsGpuEnsureCapacity = true;
    i = yy->data[p].f1->size[0] * yy->data[p].f1->size[1];
    yy->data[p].f1->size[0] = cpu_y->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(yy->data[p].f1, i, &i_emlrtRTEI);
    i = yy->data[p].f1->size[0] * yy->data[p].f1->size[1];
    yy->data[p].f1->size[1] = cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(yy->data[p].f1, i, &i_emlrtRTEI);
    profileLoopStart("ec_wtc_fp_loop_3", __LINE__,
                     (cpu_y->size[0] * cpu_y->size[1] - 1) + 1, "");
    for (i = 0; i < cpu_y->size[0] * cpu_y->size[1]; i++) {
      if (yp_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &b_gpu_y);
      }
      yp_needsGpuEnsureCapacity = false;
      yy->data[p].f1->data[i] = cpu_y->data[i];
    }
    profileLoopEnd();
    i = xx->data[p].f1->size[0] * xx->data[p].f1->size[1];
    xx->data[p].f1->size[0] = cpu_dv->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(xx->data[p].f1, i, &j_emlrtRTEI);
    i = xx->data[p].f1->size[0] * xx->data[p].f1->size[1];
    xx->data[p].f1->size[1] = cpu_dv->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(xx->data[p].f1, i, &j_emlrtRTEI);
    profileLoopStart("ec_wtc_fp_loop_4", __LINE__,
                     (cpu_dv->size[0] * cpu_dv->size[1] - 1) + 1, "");
    for (i = 0; i < cpu_dv->size[0] * cpu_dv->size[1]; i++) {
      if (dv_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
      }
      dv_outdatedOnCpu = false;
      xx->data[p].f1->data[i] = cpu_dv->data[i];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&c_cpu_yp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_r);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_w);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&yCol);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_y1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_uint32_T#" MW_AT_LINE);
  emxFree_uint32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_h);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_h1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv3);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_opts_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_opts_filter);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_yp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_yp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_x2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_x1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xp);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_yp);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_yp);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_opts_filter);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_opts_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv3);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_h1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_h);
  nvtxMarkA("#gpuEmxFree_uint32_T#" MW_AT_LINE);
  gpuEmxFree_uint32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_y1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_w);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&c_gpu_yp);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_wtc_fp.cu)

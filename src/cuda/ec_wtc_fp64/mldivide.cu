#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mldivide.cu
//
// Code generation for function 'mldivide'
//

// Include files
#include "mldivide.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "lapacke.h"
#include "hip/hip_math_constants.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstddef>

// Variable Definitions
static emlrtRTEInfo cf_emlrtRTEI{
    20,                                                              // lineNo
    5,                                                               // colNo
    "mldivide",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/mldivide.m" // pName
};

static emlrtRTEInfo
    df_emlrtRTEI{
        1,        // lineNo
        32,       // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo ef_emlrtRTEI{
    1,             // lineNo
    42,            // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

static emlrtRTEInfo
    ff_emlrtRTEI{
        61,       // lineNo
        9,        // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo gf_emlrtRTEI{
    27,            // lineNo
    23,            // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

static emlrtRTEInfo
    hf_emlrtRTEI{
        92,       // lineNo
        22,       // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo
    if_emlrtRTEI{
        105,      // lineNo
        1,        // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo jf_emlrtRTEI{
    68,            // lineNo
    9,             // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

static emlrtRTEInfo kf_emlrtRTEI{
    85,        // lineNo
    26,        // colNo
    "qrsolve", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/qrsolve.m" // pName
};

static emlrtRTEInfo lf_emlrtRTEI{
    119,       // lineNo
    5,         // colNo
    "qrsolve", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/qrsolve.m" // pName
};

static emlrtRTEInfo mf_emlrtRTEI{
    1,                                                               // lineNo
    14,                                                              // colNo
    "mldivide",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/mldivide.m" // pName
};

static emlrtRTEInfo nf_emlrtRTEI{
    27,            // lineNo
    1,             // colNo
    "xgetrfs_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrfs_gpu.m" // pName
};

// Function Declarations
static void cusolverCheck(hipsolverStatus_t errCode, const char_T *file,
                          uint32_T b_line);

static __global__ void mldivide_kernel29(const int32_T b, const int32_T na,
                                         emxArray_real_T A, int32_T A_dim0);

static __global__ void mldivide_kernel30(const emxArray_real_T B,
                                         const emxArray_int32_T jpvt,
                                         const int32_T na, emxArray_real_T Y);

// Function Definitions
static void cusolverCheck(hipsolverStatus_t errCode, const char_T *file,
                          uint32_T b_line)
{
  const char *errName;
  const char *errString;
  nvtxRangePushA("#fcn#cusolverCheck#" MW_AT_LOCATION);
  if (errCode != HIPSOLVER_STATUS_SUCCESS) {
    cusolverGetErrorName(errCode, &errName);
    cusolverGetErrorString(errCode, &errString);
    nvtxMarkA("#raiseCudaError#" MW_AT_LINE);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
  nvtxRangePop();
}

static __global__ __launch_bounds__(1024, 1) void mldivide_kernel29(
    const int32_T b, const int32_T na, emxArray_real_T A, int32_T A_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(na) + 1UL) * (static_cast<uint64_T>(b) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b) + 1UL));
    A.data[j * A_dim0 + i] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void mldivide_kernel30(
    const emxArray_real_T B, const emxArray_int32_T jpvt, const int32_T na,
    emxArray_real_T Y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(na);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    Y.data[jpvt.data[i] - 1] = B.data[i];
  }
}

//
//
namespace coder {
void mldivide(const emxArray_real_T *A, const emxArray_real_T *B,
              emxArray_real_T *cpu_Y, boolean_T *Y_outdatedOnCpu,
              emxArray_real_T *gpu_Y, boolean_T *Y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_IPIV;
  emxArray_int32_T gpu_jpvt;
  emxArray_int32_T *cpu_IPIV;
  emxArray_int32_T *cpu_jpvt;
  emxArray_ptrdiff_t *jpvt_t;
  emxArray_real_T b_gpu_A;
  emxArray_real_T gpu_A;
  emxArray_real_T gpu_B;
  emxArray_real_T gpu_tau;
  emxArray_real_T *b_cpu_A;
  emxArray_real_T *cpu_A;
  emxArray_real_T *cpu_B;
  emxArray_real_T *cpu_tau;
  int32_T cpu_maxmn;
  int32_T *gpu_maxmn;
  nvtxRangePushA("#fcn#mldivide#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_maxmn, 4UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_B);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_IPIV);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_tau);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_jpvt);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_A);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_A, 2, &mf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_jpvt, 2, &mf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_tau, 1, &mf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_A, 2, &mf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_ptrdiff_t#" MW_AT_LINE);
  emxInit_ptrdiff_t(&jpvt_t, 1, &if_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_IPIV, 1, &nf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_B, 1, &kf_emlrtRTEI, true);
  if ((A->size[0] == 0) || (A->size[1] == 0) || (B->size[0] == 0)) {
    int32_T u0;
    *Y_outdatedOnCpu = false;
    *Y_outdatedOnGpu = false;
    u0 = cpu_Y->size[0];
    cpu_Y->size[0] = A->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_Y, u0, &cf_emlrtRTEI);
    profileLoopStart("mldivide_loop_0", __LINE__, (A->size[1] - 1) + 1, "");
    for (u0 = 0; u0 < A->size[1]; u0++) {
      cpu_Y->data[u0] = 0.0;
      *Y_outdatedOnGpu = true;
    }
    profileLoopEnd();
  } else if (A->size[0] == A->size[1]) {
    int32_T minmn;
    int32_T u0;
    boolean_T A_outdatedOnGpu;
    *Y_outdatedOnCpu = false;
    *Y_outdatedOnGpu = false;
    u0 = cpu_Y->size[0];
    cpu_Y->size[0] = B->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_Y, u0, &cf_emlrtRTEI);
    profileLoopStart("mldivide_loop_2", __LINE__, (B->size[0] - 1) + 1, "");
    for (u0 = 0; u0 < B->size[0]; u0++) {
      cpu_Y->data[u0] = B->data[u0];
      *Y_outdatedOnGpu = true;
    }
    profileLoopEnd();
    u0 = A->size[0];
    minmn = A->size[1];
    if (u0 <= minmn) {
      minmn = u0;
    }
    u0 = B->size[0];
    if (u0 <= minmn) {
      minmn = u0;
    }
    A_outdatedOnGpu = false;
    u0 = b_cpu_A->size[0] * b_cpu_A->size[1];
    b_cpu_A->size[0] = A->size[0];
    b_cpu_A->size[1] = A->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b_cpu_A, u0, &ef_emlrtRTEI);
    profileLoopStart("mldivide_loop_3", __LINE__,
                     (A->size[0] * A->size[1] - 1) + 1, "");
    for (u0 = 0; u0 < A->size[0] * A->size[1]; u0++) {
      b_cpu_A->data[u0] = A->data[u0];
      A_outdatedOnGpu = true;
    }
    profileLoopEnd();
    u0 = cpu_IPIV->size[0];
    cpu_IPIV->size[0] = minmn;
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_IPIV, u0, &gf_emlrtRTEI);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(b_cpu_A, &gpu_A, !A_outdatedOnGpu);
    if (A_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_A, b_cpu_A);
    }
    nvtxMarkA("#cusolverCheck#" MW_AT_LINE);
    cusolverCheck(hipsolverDnDgetrf_bufferSize(getCuSolverGlobalHandle(), minmn,
                                              minmn, (double *)&gpu_A.data[0],
                                              b_cpu_A->size[0],
                                              getCuSolverWorkspaceReq()),
                  __FILE__, __LINE__);
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_IPIV, &gpu_IPIV, true);
    nvtxMarkA("#cusolverCheck#" MW_AT_LINE);
    cusolverCheck(
        hipsolverDnDgetrf(getCuSolverGlobalHandle(), minmn, minmn,
                         (double *)&gpu_A.data[0], b_cpu_A->size[0],
                         static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                         &gpu_IPIV.data[0], gpu_maxmn),
        __FILE__, __LINE__);
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(&cpu_maxmn, gpu_maxmn, 4UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    if (cpu_maxmn < 0) {
      *Y_outdatedOnGpu = false;
      u0 = cpu_Y->size[0];
      cpu_Y->size[0] = B->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_Y, u0, &jf_emlrtRTEI);
      profileLoopStart("mldivide_loop_6", __LINE__, (B->size[0] - 1) + 1, "");
      for (u0 = 0; u0 < B->size[0]; u0++) {
        cpu_Y->data[u0] = rtNaN;
        *Y_outdatedOnGpu = true;
      }
      profileLoopEnd();
    } else {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_Y, gpu_Y, !*Y_outdatedOnGpu);
      if (*Y_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(gpu_Y, cpu_Y);
      }
      nvtxMarkA("#cusolverCheck#" MW_AT_LINE);
      cusolverCheck(hipsolverDnDgetrs(getCuSolverGlobalHandle(), HIPBLAS_OP_N,
                                     minmn, 1, (double *)&gpu_A.data[0],
                                     b_cpu_A->size[0], &gpu_IPIV.data[0],
                                     (double *)&gpu_Y->data[0], B->size[0],
                                     gpu_maxmn),
                    __FILE__, __LINE__);
      *Y_outdatedOnGpu = false;
      *Y_outdatedOnCpu = true;
    }
  } else {
    ptrdiff_t info_t;
    real_T tol;
    int32_T minmana;
    int32_T minmn;
    int32_T na;
    int32_T u0;
    boolean_T A_outdatedOnCpu;
    boolean_T A_outdatedOnGpu;
    boolean_T B_outdatedOnGpu;
    boolean_T jpvt_outdatedOnGpu;
    boolean_T validLaunchParams;
    A_outdatedOnCpu = false;
    u0 = cpu_A->size[0] * cpu_A->size[1];
    cpu_A->size[0] = A->size[0];
    cpu_A->size[1] = A->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_A, u0, &df_emlrtRTEI);
    profileLoopStart("mldivide_loop_1", __LINE__,
                     (A->size[0] * A->size[1] - 1) + 1, "");
    for (u0 = 0; u0 < A->size[0] * A->size[1]; u0++) {
      cpu_A->data[u0] = A->data[u0];
    }
    profileLoopEnd();
    cpu_maxmn = cpu_A->size[0];
    na = cpu_A->size[1] - 1;
    jpvt_outdatedOnGpu = false;
    u0 = cpu_jpvt->size[0] * cpu_jpvt->size[1];
    cpu_jpvt->size[0] = 1;
    cpu_jpvt->size[1] = cpu_A->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_jpvt, u0, &ff_emlrtRTEI);
    u0 = cpu_A->size[0];
    minmana = cpu_A->size[1];
    if (u0 <= minmana) {
      minmana = u0;
    }
    u0 = cpu_tau->size[0];
    cpu_tau->size[0] = minmana;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_tau, u0, &hf_emlrtRTEI);
    u0 = jpvt_t->size[0];
    jpvt_t->size[0] = cpu_A->size[1];
    nvtxMarkA("#emxEnsureCapacity_ptrdiff_t#" MW_AT_LINE);
    emxEnsureCapacity_ptrdiff_t(jpvt_t, u0, &if_emlrtRTEI);
    profileLoopStart("mldivide_loop_4", __LINE__, (cpu_A->size[1] - 1) + 1, "");
    for (u0 = 0; u0 < cpu_A->size[1]; u0++) {
      cpu_jpvt->data[u0] = 0;
      jpvt_outdatedOnGpu = true;
      jpvt_t->data[u0] = (ptrdiff_t)0;
    }
    profileLoopEnd();
    info_t = LAPACKE_dgeqp3(102, (ptrdiff_t)cpu_A->size[0],
                            (ptrdiff_t)cpu_A->size[1], &cpu_A->data[0],
                            (ptrdiff_t)cpu_A->size[0], &jpvt_t->data[0],
                            &cpu_tau->data[0]);
    A_outdatedOnGpu = true;
    if ((int32_T)info_t != 0) {
      int64_T b;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(na, cpu_maxmn - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_A, &b_gpu_A, false);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_A, cpu_A);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#mldivide_kernel29#" MW_AT_LINE);
        mldivide_kernel29<<<grid, block>>>(cpu_maxmn - 1, na, b_gpu_A,
                                           cpu_A->size[0U]);
      }
      A_outdatedOnGpu = false;
      A_outdatedOnCpu = true;
      u0 = cpu_A->size[0];
      minmn = cpu_A->size[1];
      if (u0 <= minmn) {
        minmn = u0;
      }
      profileLoopStart("mldivide_loop_7", __LINE__, (minmn - 1) + 1, "");
      for (cpu_maxmn = 0; cpu_maxmn < minmn; cpu_maxmn++) {
        cpu_tau->data[cpu_maxmn] = rtNaN;
      }
      profileLoopEnd();
      nvtxMarkA("#computeEndIdx#" MW_AT_LINE);
      b = computeEndIdx(static_cast<int64_T>(minmn + 1),
                        static_cast<int64_T>(minmana), 1L);
      profileLoopStart("mldivide_loop_8", __LINE__, b + 1L, "");
      for (int64_T k{0L}; k <= b; k++) {
        cpu_tau->data[static_cast<int32_T>((minmn + 1) + k) - 1] = 0.0;
      }
      profileLoopEnd();
      profileLoopStart("mldivide_loop_10", __LINE__, na + 1, "");
      for (cpu_maxmn = 0; cpu_maxmn <= na; cpu_maxmn++) {
        cpu_jpvt->data[cpu_maxmn] = cpu_maxmn + 1;
        jpvt_outdatedOnGpu = true;
      }
      profileLoopEnd();
    } else {
      profileLoopStart("mldivide_loop_5", __LINE__, na + 1, "");
      for (cpu_maxmn = 0; cpu_maxmn <= na; cpu_maxmn++) {
        cpu_jpvt->data[cpu_maxmn] = (int32_T)jpvt_t->data[cpu_maxmn];
        jpvt_outdatedOnGpu = true;
      }
      profileLoopEnd();
    }
    na = -1;
    if (cpu_A->size[0] < cpu_A->size[1]) {
      minmn = cpu_A->size[0];
      cpu_maxmn = cpu_A->size[1];
    } else {
      minmn = cpu_A->size[1];
      cpu_maxmn = cpu_A->size[0];
    }
    if (A_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_A, &b_gpu_A);
    }
    tol = std::fmin(1.4901161193847656E-8,
                    2.2204460492503131E-15 * static_cast<real_T>(cpu_maxmn)) *
          std::abs(cpu_A->data[0]);
    nvtxRangePushA("#loop#mldivide_whileloop_0##" MW_AT_LINE);
    while ((na + 1 < minmn) &&
           (!(std::abs(cpu_A->data[(na + cpu_A->size[0] * (na + 1)) + 1]) <=
              tol))) {
      na++;
    }
    nvtxRangePop();
    B_outdatedOnGpu = false;
    u0 = cpu_B->size[0];
    cpu_B->size[0] = B->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_B, u0, &kf_emlrtRTEI);
    profileLoopStart("mldivide_loop_9", __LINE__, (B->size[0] - 1) + 1, "");
    for (u0 = 0; u0 < B->size[0]; u0++) {
      cpu_B->data[u0] = B->data[u0];
      B_outdatedOnGpu = true;
    }
    profileLoopEnd();
    *Y_outdatedOnGpu = false;
    u0 = cpu_Y->size[0];
    cpu_Y->size[0] = cpu_A->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_Y, u0, &cf_emlrtRTEI);
    profileLoopStart("mldivide_loop_11", __LINE__, (cpu_A->size[1] - 1) + 1,
                     "");
    for (u0 = 0; u0 < cpu_A->size[1]; u0++) {
      cpu_Y->data[u0] = 0.0;
      *Y_outdatedOnGpu = true;
    }
    profileLoopEnd();
    cpu_maxmn = 0;
    u0 = cpu_A->size[0];
    minmn = cpu_A->size[1];
    if (u0 <= minmn) {
      minmn = u0;
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_A, &b_gpu_A, !A_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_tau, &gpu_tau, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_B, &gpu_B, !B_outdatedOnGpu);
    if (A_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_A, cpu_A);
    }
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_tau, cpu_tau);
    if (B_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_B, cpu_B);
    }
    nvtxMarkA("#cusolverCheck#" MW_AT_LINE);
    cusolverCheck(hipsolverDnDormqr_bufferSize(
                      getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                      cpu_B->size[0], 1, minmn, (double *)&b_gpu_A.data[0],
                      cpu_A->size[0], (double *)&gpu_tau.data[0],
                      (double *)&gpu_B.data[0], cpu_B->size[0],
                      getCuSolverWorkspaceReq()),
                  __FILE__, __LINE__);
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    A_outdatedOnCpu = false;
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(gpu_maxmn, &cpu_maxmn, 4UL, hipMemcpyHostToDevice),
        __FILE__, __LINE__);
    nvtxMarkA("#cusolverCheck#" MW_AT_LINE);
    cusolverCheck(hipsolverDnDormqr(
                      getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                      cpu_B->size[0], 1, minmn, (double *)&b_gpu_A.data[0],
                      cpu_A->size[0], (double *)&gpu_tau.data[0],
                      (double *)&gpu_B.data[0], cpu_B->size[0],
                      static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                      *getCuSolverWorkspaceReq(), gpu_maxmn),
                  __FILE__, __LINE__);
    B_outdatedOnGpu = false;
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(&cpu_maxmn, gpu_maxmn, 4UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    if (cpu_maxmn < 0) {
      cpu_maxmn = cpu_B->size[0];
      u0 = cpu_B->size[0];
      cpu_B->size[0] = cpu_maxmn;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_B, u0, &lf_emlrtRTEI);
      A_outdatedOnCpu = true;
      profileLoopStart("mldivide_loop_12", __LINE__, (cpu_maxmn - 1) + 1, "");
      for (u0 = 0; u0 < cpu_maxmn; u0++) {
        cpu_B->data[u0] = rtNaN;
        B_outdatedOnGpu = true;
      }
      profileLoopEnd();
    }
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (A_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_B, &gpu_B, !B_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_jpvt, &gpu_jpvt, !jpvt_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_Y, gpu_Y, !*Y_outdatedOnGpu);
    if (B_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_B, cpu_B);
    }
    if (jpvt_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_jpvt, cpu_jpvt);
    }
    if (*Y_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_Y, cpu_Y);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#mldivide_kernel30#" MW_AT_LINE);
      mldivide_kernel30<<<grid, block>>>(gpu_B, gpu_jpvt, na, *gpu_Y);
    }
    *Y_outdatedOnGpu = false;
    *Y_outdatedOnCpu = true;
    profileLoopStart("mldivide_loop_13", __LINE__, na + 1, "");
    for (minmn = 0; minmn <= na; minmn++) {
      u0 = cpu_jpvt->data[na - minmn];
      if (*Y_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_Y, gpu_Y);
      }
      cpu_Y->data[u0 - 1] /=
          cpu_A->data[(na - minmn) + cpu_A->size[0] * (na - minmn)];
      *Y_outdatedOnCpu = false;
      *Y_outdatedOnGpu = true;
      cpu_maxmn = na - minmn;
      profileLoopStart("mldivide_loop_14", __LINE__, (cpu_maxmn - 1) + 1, "");
      for (u0 = 0; u0 < cpu_maxmn; u0++) {
        cpu_Y->data[cpu_jpvt->data[u0] - 1] -=
            cpu_Y->data[cpu_jpvt->data[na - minmn] - 1] *
            cpu_A->data[u0 + cpu_A->size[0] * (na - minmn)];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_B);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_IPIV);
  nvtxMarkA("#emxFree_ptrdiff_t#" MW_AT_LINE);
  emxFree_ptrdiff_t(&jpvt_t);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_tau);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_jpvt);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_A);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_A);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_jpvt);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_tau);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_A);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_IPIV);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_B);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_maxmn), __FILE__, __LINE__);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (mldivide.cu)

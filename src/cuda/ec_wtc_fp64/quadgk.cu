#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// quadgk.cu
//
// Code generation for function 'quadgk'
//

// Include files
#include "quadgk.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "morseproperties.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo id_emlrtRTEI{
    63,                // lineNo
    57,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo kd_emlrtRTEI{
    63,                // lineNo
    60,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo ld_emlrtRTEI{
    1,                                                                // lineNo
    23,                                                               // colNo
    "quadgk",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/funfun/quadgk.m" // pName
};

static emlrtRTEInfo md_emlrtRTEI{
    63,                // lineNo
    16,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo nd_emlrtRTEI{
    63,                // lineNo
    20,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo od_emlrtRTEI{
    63,                // lineNo
    34,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

// Function Definitions
void binary_expand_op_3(emxArray_real_T *in1, const emxArray_real_T *in2,
                        const emxArray_real_T *in3, const emxArray_real_T *in4)
{
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  nvtxRangePushA("#fcn#binary_expand_op_3#" MW_AT_LOCATION);
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  in1->size[1] = in4->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(in1, i, &cd_emlrtRTEI);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_1 = (in3->size[1] != 1);
  profileLoopStart("binary_expand_op_3_loop_0", __LINE__,
                   (in4->size[1] - 1) + 1, "");
  for (i = 0; i < in4->size[1]; i++) {
    in1->data[i] =
        in2->data[i * stride_0_1] * in3->data[i * stride_1_1] * in4->data[i];
  }
  profileLoopEnd();
  nvtxRangePop();
}

//
//
namespace coder {
real_T quadgk(real_T fun_workspace_be, real_T fun_workspace_ga)
{
  emxArray_real_T *a;
  emxArray_real_T *b_dv1;
  emxArray_real_T *b_x;
  emxArray_real_T *b_y;
  emxArray_real_T *c_y;
  emxArray_real_T *d_y;
  emxArray_real_T *fx;
  emxArray_real_T *x;
  emxArray_real_T *xt;
  emxArray_real_T *y;
  real_T subs[1298];
  real_T interval[650];
  real_T errsub[649];
  real_T qsub[649];
  real_T pathlen;
  real_T q;
  int32_T ix;
  nvtxRangePushA("#fcn#quadgk#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  interval[0] = 0.0;
  interval[1] = 1.0;
  std::memset(&interval[2], 0, 648U * sizeof(real_T));
  q = 0.0;
  nvtxMarkA("#split#" MW_AT_LINE);
  ix = split(interval, 2, &pathlen);
  if (!(pathlen > 0.0)) {
    real_T midpt;
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    midpt = fun_workspace_be * rt_powd_snf(rtInf, fun_workspace_be - 1.0) -
            fun_workspace_ga *
                rt_powd_snf(rtInf, (fun_workspace_be + fun_workspace_ga) - 1.0);
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    q = rtInf *
        (midpt * midpt * std::exp(-2.0 * rt_powd_snf(rtInf, fun_workspace_ga)));
  } else {
    real_T err_ok;
    real_T q_ok;
    int32_T k;
    int32_T nsubs;
    boolean_T first_iteration;
    nsubs = ix - 2;
    profileLoopStart("quadgk_loop_0", __LINE__, nsubs + 1, "");
    for (k = 0; k <= nsubs; k++) {
      subs[k << 1] = interval[k];
      subs[(k << 1) + 1] = interval[k + 1];
    }
    profileLoopEnd();
    q_ok = 0.0;
    err_ok = 0.0;
    first_iteration = true;
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&x, 2, &wc_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_x, 2, &dd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&xt, 2, &ed_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&y, 2, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&fx, 2, &ld_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_dv1, 2, &md_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_y, 2, &nd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_y, 2, &od_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&d_y, 2, &md_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&a, 2, &jd_emlrtRTEI, true);
    nvtxRangePushA("#loop#quadgk_whileloop_0##" MW_AT_LINE);
    int32_T exitg1;
    do {
      real_T d;
      real_T halfh;
      real_T midpt;
      real_T tau;
      boolean_T guard1;
      exitg1 = 0;
      ix = x->size[0] * x->size[1];
      x->size[0] = 1;
      x->size[1] = 15 * (nsubs + 1);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(x, ix, &wc_emlrtRTEI);
      ix = -1;
      profileLoopStart("quadgk_loop_1", __LINE__, nsubs + 1, "");
      for (k = 0; k <= nsubs; k++) {
        d = subs[k << 1];
        tau = subs[(k << 1) + 1];
        midpt = (d + tau) / 2.0;
        halfh = (tau - d) / 2.0;
        profileLoopStart("quadgk_loop_2", __LINE__, 14 + 1, "");
        for (int32_T j{0}; j < 15; j++) {
          x->data[(ix + j) + 1] = dv[j] * halfh + midpt;
        }
        profileLoopEnd();
        ix += 15;
      }
      profileLoopEnd();
      ix = b_x->size[0] * b_x->size[1];
      b_x->size[0] = 1;
      b_x->size[1] = x->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_x, ix, &xc_emlrtRTEI);
      ix = xt->size[0] * xt->size[1];
      xt->size[0] = 1;
      xt->size[1] = x->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(xt, ix, &yc_emlrtRTEI);
      ix = x->size[1];
      profileLoopStart("quadgk_loop_3", __LINE__, (ix - 1) + 1, "");
      for (k = 0; k < ix; k++) {
        d = x->data[k];
        midpt = d / (1.0 - d);
        b_x->data[k] = midpt * midpt;
        xt->data[k] = 2.0 * midpt / ((1.0 - d) * (1.0 - d));
      }
      profileLoopEnd();
      guard1 = false;
      if (!first_iteration) {
        boolean_T exitg2;
        midpt = std::abs(b_x->data[0]);
        k = 0;
        exitg2 = false;
        nvtxRangePushA("#loop#quadgk_whileloop_1##" MW_AT_LINE);
        while ((!exitg2) && (k <= b_x->size[1] - 2)) {
          tau = midpt;
          midpt = std::abs(b_x->data[k + 1]);
          if (std::abs(b_x->data[k + 1] - b_x->data[k]) <=
              2.2204460492503131E-14 * std::fmax(tau, midpt)) {
            first_iteration = true;
            exitg2 = true;
          } else {
            k++;
          }
        }
        nvtxRangePop();
        if (first_iteration) {
          int16_T dv_idx_1;
          dv_idx_1 = static_cast<int16_T>(x->size[1]);
          ix = fx->size[0] * fx->size[1];
          fx->size[0] = 1;
          fx->size[1] = x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(fx, ix, &bd_emlrtRTEI);
          profileLoopStart("quadgk_loop_7", __LINE__, (dv_idx_1 - 1) + 1, "");
          for (ix = 0; ix < dv_idx_1; ix++) {
            fx->data[ix] = 0.0;
          }
          profileLoopEnd();
        } else {
          guard1 = true;
        }
      } else {
        guard1 = true;
      }
      if (guard1) {
        first_iteration = false;
        ix = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = b_x->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, ix, &kb_emlrtRTEI);
        ix = b_x->size[1];
        profileLoopStart("quadgk_loop_5", __LINE__, (ix - 1) + 1, "");
        for (k = 0; k < ix; k++) {
          nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
          y->data[k] = rt_powd_snf(b_x->data[k], fun_workspace_ga);
        }
        profileLoopEnd();
        ix = b_dv1->size[0] * b_dv1->size[1];
        b_dv1->size[0] = 1;
        b_dv1->size[1] = y->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_dv1, ix, &id_emlrtRTEI);
        profileLoopStart("quadgk_loop_9", __LINE__, (y->size[1] - 1) + 1, "");
        for (ix = 0; ix < y->size[1]; ix++) {
          b_dv1->data[ix] = -2.0 * y->data[ix];
        }
        profileLoopEnd();
        ix = b_x->size[1];
        profileLoopStart("quadgk_loop_10", __LINE__, (ix - 1) + 1, "");
        for (k = 0; k < ix; k++) {
          b_dv1->data[k] = std::exp(b_dv1->data[k]);
        }
        profileLoopEnd();
        ix = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = b_x->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_y, ix, &kb_emlrtRTEI);
        ix = b_x->size[1];
        profileLoopStart("quadgk_loop_12", __LINE__, (ix - 1) + 1, "");
        for (k = 0; k < ix; k++) {
          nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
          b_y->data[k] = rt_powd_snf(b_x->data[k], fun_workspace_be - 1.0);
        }
        profileLoopEnd();
        midpt = (fun_workspace_be + fun_workspace_ga) - 1.0;
        ix = c_y->size[0] * c_y->size[1];
        c_y->size[0] = 1;
        c_y->size[1] = b_x->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_y, ix, &kb_emlrtRTEI);
        ix = b_x->size[1];
        profileLoopStart("quadgk_loop_13", __LINE__, (ix - 1) + 1, "");
        for (k = 0; k < ix; k++) {
          nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
          c_y->data[k] = rt_powd_snf(b_x->data[k], midpt);
        }
        profileLoopEnd();
        if (b_y->size[1] == c_y->size[1]) {
          ix = a->size[0] * a->size[1];
          a->size[0] = 1;
          a->size[1] = b_y->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(a, ix, &jd_emlrtRTEI);
          profileLoopStart("quadgk_loop_14", __LINE__, (b_y->size[1] - 1) + 1,
                           "");
          for (ix = 0; ix < b_y->size[1]; ix++) {
            a->data[ix] = fun_workspace_be * b_y->data[ix] -
                          fun_workspace_ga * c_y->data[ix];
          }
          profileLoopEnd();
        } else {
          nvtxMarkA("#binary_expand_op_5#" MW_AT_LINE);
          binary_expand_op_5(a, fun_workspace_be, b_y, fun_workspace_ga, c_y);
        }
        ix = d_y->size[0] * d_y->size[1];
        d_y->size[0] = 1;
        d_y->size[1] = a->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(d_y, ix, &kb_emlrtRTEI);
        ix = a->size[1];
        profileLoopStart("quadgk_loop_15", __LINE__, (ix - 1) + 1, "");
        for (k = 0; k < ix; k++) {
          d = a->data[k];
          d_y->data[k] = d * d;
        }
        profileLoopEnd();
        if (d_y->size[1] == b_dv1->size[1]) {
          ix = fx->size[0] * fx->size[1];
          fx->size[0] = 1;
          fx->size[1] = d_y->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(fx, ix, &cd_emlrtRTEI);
          profileLoopStart("quadgk_loop_16", __LINE__, (d_y->size[1] - 1) + 1,
                           "");
          for (ix = 0; ix < d_y->size[1]; ix++) {
            fx->data[ix] = d_y->data[ix] * b_dv1->data[ix] * xt->data[ix];
          }
          profileLoopEnd();
        } else {
          nvtxMarkA("#binary_expand_op_3#" MW_AT_LINE);
          binary_expand_op_3(fx, d_y, b_dv1, xt);
        }
      }
      if (first_iteration) {
        exitg1 = 1;
      } else {
        real_T tol;
        midpt = 0.0;
        ix = -1;
        profileLoopStart("quadgk_loop_4", __LINE__, nsubs + 1, "");
        for (k = 0; k <= nsubs; k++) {
          d = 0.0;
          tau = 0.0;
          profileLoopStart("quadgk_loop_6", __LINE__, 14 + 1, "");
          for (int32_T j{0}; j < 15; j++) {
            d += dv1[j] * fx->data[(ix + j) + 1];
            tau += dv2[j] * fx->data[(ix + j) + 1];
          }
          profileLoopEnd();
          ix += 15;
          halfh = (subs[(k << 1) + 1] - subs[k << 1]) / 2.0;
          d *= halfh;
          qsub[k] = d;
          midpt += d;
          errsub[k] = tau * halfh;
        }
        profileLoopEnd();
        q = midpt + q_ok;
        tol = std::fmax(1.0E-10, 1.0E-6 * std::abs(q));
        tau = 2.0 * tol / pathlen;
        midpt = 0.0;
        ix = 0;
        profileLoopStart("quadgk_loop_8", __LINE__, nsubs + 1, "");
        for (k = 0; k <= nsubs; k++) {
          d = errsub[k];
          halfh = std::abs(d);
          if (halfh <= tau * ((subs[(k << 1) + 1] - subs[k << 1]) / 2.0)) {
            err_ok += d;
            q_ok += qsub[k];
          } else {
            midpt += halfh;
            ix++;
            subs[(ix - 1) << 1] = subs[k << 1];
            subs[((ix - 1) << 1) + 1] = subs[(k << 1) + 1];
          }
        }
        profileLoopEnd();
        midpt += std::abs(err_ok);
        if ((!std::isinf(q)) && (!std::isnan(q)) &&
            ((!std::isinf(midpt)) && (!std::isnan(midpt))) && (ix != 0) &&
            (!(midpt <= tol))) {
          nsubs = (ix << 1) - 1;
          if (nsubs + 1 > 650) {
            exitg1 = 1;
          } else {
            profileLoopStart("quadgk_loop_11", __LINE__, (ix - 1) + 1, "");
            for (k = 0; k < ix; k++) {
              subs[((((ix - k) << 1) - 1) << 1) + 1] =
                  subs[(((ix - k) - 1) << 1) + 1];
              subs[(((ix - k) << 1) - 1) << 1] =
                  (subs[((ix - k) - 1) << 1] +
                   subs[(((ix - k) - 1) << 1) + 1]) /
                  2.0;
              subs[((((ix - k) << 1) - 2) << 1) + 1] =
                  subs[(((ix - k) << 1) - 1) << 1];
              subs[(((ix - k) << 1) - 2) << 1] = subs[((ix - k) - 1) << 1];
            }
            profileLoopEnd();
          }
        } else {
          exitg1 = 1;
        }
      }
    } while (exitg1 == 0);
    nvtxRangePop();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&a);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&d_y);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_y);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_y);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_dv1);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&fx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&y);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&xt);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_x);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&x);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
  return q;
}

//
//
int32_T split(real_T x[650], int32_T nx, real_T *pathlen)
{
  int32_T n_idx_0;
  int32_T nxnew;
  nvtxRangePushA("#fcn#split#" MW_AT_LOCATION);
  *pathlen = x[1] - x[0];
  if (*pathlen > 0.0) {
    real_T udelta;
    udelta = 10.0 / *pathlen;
    n_idx_0 =
        static_cast<int32_T>(std::ceil(std::abs(x[1] - x[0]) * udelta) - 1.0);
    nxnew =
        static_cast<int32_T>(std::ceil(std::abs(x[1] - x[0]) * udelta) - 1.0) +
        2;
    if (nxnew > 2) {
      x[nxnew - 1] = x[1];
      udelta = (x[1] - x[0]) / static_cast<real_T>(n_idx_0 + 1);
      profileLoopStart("split_loop_1", __LINE__, (n_idx_0 - 1) + 1, "");
      for (int32_T j{0}; j < n_idx_0; j++) {
        x[(nxnew - j) - 2] = x[0] + static_cast<real_T>(n_idx_0 - j) * udelta;
      }
      profileLoopEnd();
    }
    nx = nxnew;
  } else {
    nxnew = 2;
  }
  n_idx_0 = 0;
  profileLoopStart("split_loop_0", __LINE__, (nx - 2) + 1, "");
  for (int32_T j{0}; j <= nx - 2; j++) {
    if (std::abs(x[j + 1] - x[n_idx_0]) > 0.0) {
      n_idx_0++;
      x[n_idx_0] = x[j + 1];
    } else {
      nxnew--;
    }
  }
  profileLoopEnd();
  if (nxnew < 2) {
    x[1] = x[nx - 1];
    nxnew = 2;
  }
  nvtxRangePop();
  return nxnew;
}

} // namespace coder

// End of code generation (quadgk.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wtc_fp_initialize.cu
//
// Code generation for function 'ec_wtc_fp_initialize'
//

// Include files
#include "ec_wtc_fp_initialize.h"
#include "_coder_ec_wtc_fp_mex.h"
#include "ec_wtc_fp_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void ec_wtc_fp_once();

// Function Definitions
static void ec_wtc_fp_once()
{
  mex_InitInfAndNan();
  nvtxRangePushA("#initFcn#ec_wtc_fp_once#" MW_AT_LOCATION);
  // Initialize GPU by calling hipFree(nullptr)
  nvtxMarkA("#hipFree#" MW_AT_LINE);
  hipFree(nullptr);
  nvtxRangePop();
}

void ec_wtc_fp_initialize()
{
  nvtxRangePushA("#fcn#ec_wtc_fp_initialize#" MW_AT_LOCATION);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "wavelet_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "signal_toolbox", 2);
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  cusolverEnsureInitialization();
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    nvtxMarkA("#ec_wtc_fp_once#" MW_AT_LINE);
    ec_wtc_fp_once();
  }
  nvtxRangePop();
}

// End of code generation (ec_wtc_fp_initialize.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_filtfilt_fp.cu
//
// Code generation for function 'ec_filtfilt_fp'
//

// Include files
#include "ec_filtfilt_fp.h"
#include "ec_filtfilt_fp_data.h"
#include "ec_filtfilt_fp_emxutil.h"
#include "ec_filtfilt_fp_mexutil.h"
#include "ec_filtfilt_fp_types.h"
#include "filter.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    99,                                                           // lineNo
    1,                                                            // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    103,                                                          // lineNo
    28,                                                           // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    60,                                                              // lineNo
    20,                                                              // colNo
    "bsxfun",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/bsxfun.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    103,                                                          // lineNo
    15,                                                           // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo
    e_emlrtRTEI{
        76,                  // lineNo
        13,                  // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo f_emlrtRTEI{
    103,                                                          // lineNo
    5,                                                            // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    132,                                                          // lineNo
    5,                                                            // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    1,                                                            // lineNo
    14,                                                           // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    135,                                                          // lineNo
    10,                                                           // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    136,                                                          // lineNo
    5,                                                            // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    139,                                                          // lineNo
    5,                                                            // colNo
    "ec_filtfilt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt_fp.m" // pName
};

// Function Declarations
static int32_T div_s32(int32_T numerator, int32_T denominator);

static __global__ void
ec_filtfilt_fp_kernel1(const int32_T ch, const emxArray_real32_T x,
                       const int32_T b_x, emxArray_real32_T fv, int32_T x_dim0);

static __global__ void ec_filtfilt_fp_kernel10(const emxArray_real32_T a,
                                               const int32_T b_a,
                                               emxArray_real32_T c_a);

static __global__ void ec_filtfilt_fp_kernel11(const emxArray_real32_T b,
                                               const int32_T b_b,
                                               emxArray_real32_T c_b);

static __global__ void ec_filtfilt_fp_kernel12(const emxArray_real32_T a,
                                               const int32_T b_a,
                                               emxArray_real32_T c_a);

static __global__ void ec_filtfilt_fp_kernel13(const emxArray_real32_T yc3,
                                               const int32_T bcoef,
                                               const int32_T i5,
                                               const int32_T b,
                                               emxArray_real32_T b_yc3);

static __global__ void ec_filtfilt_fp_kernel14(const real32_T a,
                                               const emxArray_real32_T z,
                                               const int32_T b_z,
                                               emxArray_real32_T c_z);

static __global__ void ec_filtfilt_fp_kernel15(const emxArray_real32_T b,
                                               const int32_T b_b,
                                               emxArray_real32_T c_b);

static __global__ void ec_filtfilt_fp_kernel16(const emxArray_real32_T a,
                                               const int32_T b_a,
                                               emxArray_real32_T c_a);

static __global__ void ec_filtfilt_fp_kernel17(const emxArray_real32_T yc2,
                                               const int32_T bcoef,
                                               const int32_T i5,
                                               const int32_T b,
                                               emxArray_real32_T b_yc2);

static __global__ void ec_filtfilt_fp_kernel18(const emxArray_real32_T yc5,
                                               const int32_T bcoef,
                                               const int32_T i5,
                                               const int32_T b,
                                               emxArray_real32_T fv);

static __global__ void
ec_filtfilt_fp_kernel19(const emxArray_real32_T fv, const int32_T ch,
                        const int32_T b, emxArray_real32_T y, int32_T y_dim0);

static __global__ void
ec_filtfilt_fp_kernel2(const emxArray_real32_T fv, const int32_T bcoef,
                       const int32_T i3, const int32_T i2, const real32_T a,
                       const int32_T i5, emxArray_real32_T xt);

static __global__ void ec_filtfilt_fp_kernel3(const emxArray_real32_T b,
                                              const int32_T b_b,
                                              emxArray_real32_T c_b);

static __global__ void ec_filtfilt_fp_kernel4(const emxArray_real32_T a,
                                              const int32_T b_a,
                                              emxArray_real32_T c_a);

static __global__ void ec_filtfilt_fp_kernel5(const real32_T a,
                                              const emxArray_real32_T z,
                                              const int32_T b_z,
                                              emxArray_real32_T c_z);

static __global__ void ec_filtfilt_fp_kernel6(const emxArray_real32_T b,
                                              const int32_T b_b,
                                              emxArray_real32_T c_b);

static __global__ void ec_filtfilt_fp_kernel7(const emxArray_real32_T a,
                                              const int32_T b_a,
                                              emxArray_real32_T c_a);

static __global__ void
ec_filtfilt_fp_kernel8(const emxArray_real32_T fv, const int32_T bcoef,
                       const int32_T i7, const int32_T i6, const real32_T a,
                       const int32_T i5, emxArray_real32_T c);

static __global__ void ec_filtfilt_fp_kernel9(const emxArray_real32_T b,
                                              const int32_T b_b,
                                              emxArray_real32_T c_b);

// Function Definitions
static int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  nvtxRangePushA("#fcn#div_s32#" MW_AT_LOCATION);
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  nvtxRangePop();
  return quotient;
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel1(
    const int32_T ch, const emxArray_real32_T x, const int32_T b_x,
    emxArray_real32_T fv, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    fv.data[k] = x.data[k + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel10(
    const emxArray_real32_T a, const int32_T b_a, emxArray_real32_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel11(
    const emxArray_real32_T b, const int32_T b_b, emxArray_real32_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel12(
    const emxArray_real32_T a, const int32_T b_a, emxArray_real32_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel13(
    const emxArray_real32_T yc3, const int32_T bcoef, const int32_T i5,
    const int32_T b, emxArray_real32_T b_yc3)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_yc3.data[k] = yc3.data[i5 + bcoef * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel14(
    const real32_T a, const emxArray_real32_T z, const int32_T b_z,
    emxArray_real32_T c_z)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_z);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_z.data[k] = z.data[k] * a;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel15(
    const emxArray_real32_T b, const int32_T b_b, emxArray_real32_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel16(
    const emxArray_real32_T a, const int32_T b_a, emxArray_real32_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel17(
    const emxArray_real32_T yc2, const int32_T bcoef, const int32_T i5,
    const int32_T b, emxArray_real32_T b_yc2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_yc2.data[k] = yc2.data[i5 + bcoef * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel18(
    const emxArray_real32_T yc5, const int32_T bcoef, const int32_T i5,
    const int32_T b, emxArray_real32_T fv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    fv.data[k] = yc5.data[i5 + bcoef * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel19(
    const emxArray_real32_T fv, const int32_T ch, const int32_T b,
    emxArray_real32_T y, int32_T y_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k + y_dim0 * ch] = fv.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel2(
    const emxArray_real32_T fv, const int32_T bcoef, const int32_T i3,
    const int32_T i2, const real32_T a, const int32_T i5, emxArray_real32_T xt)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i5);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    xt.data[k] = a - fv.data[(i2 + i3 * (bcoef * k)) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel3(
    const emxArray_real32_T b, const int32_T b_b, emxArray_real32_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel4(
    const emxArray_real32_T a, const int32_T b_a, emxArray_real32_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel5(
    const real32_T a, const emxArray_real32_T z, const int32_T b_z,
    emxArray_real32_T c_z)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_z);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_z.data[k] = z.data[k] * a;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel6(
    const emxArray_real32_T b, const int32_T b_b, emxArray_real32_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel7(
    const emxArray_real32_T a, const int32_T b_a, emxArray_real32_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel8(
    const emxArray_real32_T fv, const int32_T bcoef, const int32_T i7,
    const int32_T i6, const real32_T a, const int32_T i5, emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i5);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c.data[k] = a - fv.data[(i6 + i7 * (bcoef * k)) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt_fp_kernel9(
    const emxArray_real32_T b, const int32_T b_b, emxArray_real32_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

//
// function y = ec_filtfilt_fp(x,b,a,z,nf,L)
void ec_filtfilt_fp(const emxArray_real32_T *cpu_x,
                    const emxArray_real32_T *cpu_b,
                    const emxArray_real32_T *cpu_a,
                    const emxArray_real32_T *cpu_z, real32_T nf, real32_T L,
                    emxArray_real32_T *cpu_y)
{
  dim3 block;
  dim3 grid;
  emxArray_real32_T b_gpu_a;
  emxArray_real32_T b_gpu_b;
  emxArray_real32_T b_gpu_yc2;
  emxArray_real32_T b_gpu_yc3;
  emxArray_real32_T b_gpu_z;
  emxArray_real32_T b_gpu_zo;
  emxArray_real32_T c_gpu_a;
  emxArray_real32_T c_gpu_b;
  emxArray_real32_T c_gpu_z;
  emxArray_real32_T c_gpu_zo;
  emxArray_real32_T d_gpu_a;
  emxArray_real32_T d_gpu_b;
  emxArray_real32_T e_gpu_a;
  emxArray_real32_T e_gpu_b;
  emxArray_real32_T f_gpu_a;
  emxArray_real32_T f_gpu_b;
  emxArray_real32_T gpu__1;
  emxArray_real32_T gpu__2;
  emxArray_real32_T gpu_a;
  emxArray_real32_T gpu_b;
  emxArray_real32_T gpu_c;
  emxArray_real32_T gpu_fv;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_xt;
  emxArray_real32_T gpu_y;
  emxArray_real32_T gpu_yc2;
  emxArray_real32_T gpu_yc3;
  emxArray_real32_T gpu_yc5;
  emxArray_real32_T gpu_z;
  emxArray_real32_T gpu_zo;
  emxArray_real32_T *b_cpu_a;
  emxArray_real32_T *b_cpu_b;
  emxArray_real32_T *b_cpu_yc2;
  emxArray_real32_T *b_cpu_yc3;
  emxArray_real32_T *b_cpu_z;
  emxArray_real32_T *b_cpu_zo;
  emxArray_real32_T *c_cpu_a;
  emxArray_real32_T *c_cpu_b;
  emxArray_real32_T *c_cpu_z;
  emxArray_real32_T *c_cpu_zo;
  emxArray_real32_T *cpu__1;
  emxArray_real32_T *cpu__2;
  emxArray_real32_T *cpu_c;
  emxArray_real32_T *cpu_fv;
  emxArray_real32_T *cpu_xt;
  emxArray_real32_T *cpu_yc2;
  emxArray_real32_T *cpu_yc3;
  emxArray_real32_T *cpu_yc5;
  emxArray_real32_T *cpu_zo;
  emxArray_real32_T *d_cpu_a;
  emxArray_real32_T *d_cpu_b;
  emxArray_real32_T *e_cpu_a;
  emxArray_real32_T *e_cpu_b;
  emxArray_real32_T *f_cpu_a;
  emxArray_real32_T *f_cpu_b;
  int32_T b_i;
  int32_T csz_idx_0;
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T i3;
  int32_T i4;
  boolean_T a_outdatedOnCpu;
  boolean_T a_outdatedOnGpu;
  boolean_T b_outdatedOnCpu;
  boolean_T b_outdatedOnGpu;
  boolean_T b_zo_outdatedOnGpu;
  boolean_T fv_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T xt_outdatedOnCpu;
  boolean_T y_needsGpuEnsureCapacity;
  boolean_T yc2_outdatedOnCpu;
  boolean_T yc2_outdatedOnGpu;
  boolean_T yc3_outdatedOnCpu;
  boolean_T yc3_outdatedOnGpu;
  boolean_T yc5_outdatedOnCpu;
  boolean_T yc5_outdatedOnGpu;
  boolean_T z_needsGpuEnsureCapacity;
  boolean_T z_outdatedOnGpu;
  boolean_T zo_outdatedOnCpu;
  boolean_T zo_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_filtfilt_fp#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&f_gpu_a);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&f_gpu_b);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_z);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&e_gpu_a);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&e_gpu_b);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_a);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_b);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_a);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_b);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_z);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_b);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_yc5);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu__2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_c);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu__1);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_xt);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_fv);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_z);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_a);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_b);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_x);
  zo_outdatedOnCpu = false;
  b_zo_outdatedOnGpu = false;
  yc2_outdatedOnGpu = false;
  yc5_outdatedOnGpu = false;
  z_outdatedOnGpu = true;
  a_outdatedOnGpu = true;
  b_outdatedOnGpu = true;
  x_outdatedOnGpu = true;
  z_needsGpuEnsureCapacity = true;
  yc3_outdatedOnGpu = true;
  zo_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  [ec_filtfilt1] helper function for ec_filtfilt()
  //  FILTFILT Zero-phase forward and reverse digital IIR filtering.
  //    Y = FILTFILT(B,A,X) filters the data in vector, matrix, or N-D
  //    array, X, with the filter described by vectors A and B to create
  //    the filtered data Y. The filter is described by the difference
  //    equation:
  //
  //      a(1)*y(n) = b(1)*x(n) + b(2)*x(n-1) + ... + b(nb+1)*x(n-nb)
  //                            - a(2)*y(n-1) - ... - a(na+1)*y(n-na)
  //
  //    filtfilt operates along the first dimension of X except when X is a
  //    row vector in which case it operates along the second dimension.
  //    The length of the input channels must be more than three times the
  //    filter order, defined as max(length(B)-1,length(A)-1).
  //
  //    Y = FILTFILT(SOS,G,X) filters the data in vector, matrix, or N-D
  //    array, X, with the second-order section (SOS) filter described by
  //    the matrix SOS and the vector G. The coefficients of the SOS matrix
  //    must be expressed using an Lx6 matrix where L is the number of
  //    second-order sections. The scale values of the filter must be
  //    expressed using the vector G. The length of G must be between 1 and
  //    L+1, and the length of input channels in X must be more than three
  //    times the filter order (input channel length must be greater than
  //    one when the order is zero). You can use filtord(SOS) to get the
  //    order of the filter. The SOS matrix should have the following form:
  //
  //    SOS = [ b01 b11 b21 a01 a11 a21
  //            b02 b12 b22 a02 a12 a22
  //            ...
  //            b0L b1L b2L a0L a1L a2L ]
  //
  //    Y = FILTFILT(D,X) filters the data in vector, matrix, or N-D
  //    array, X, with the digital filter D. You design a digital filter,
  //    D, by calling the <a href="matlab:help designfilt">designfilt</a>
  //    function. The length of the input channels in X must be more than
  //    three times the filter order. You can use filtord(D) to get the
  //    order of the digital filter D.
  //
  //    After filtering in the forward direction, the filtered data is
  //    reversed and run back through the filter; Y is the time reverse of
  //    the output of the second filtering operation. The result has
  //    precisely zero phase distortion, and magnitude modified by the
  //    square of the filter's magnitude response. Startup and ending
  //    transients are minimized by matching initial conditions.
  //
  //    Note that FILTFILT should not be used when the intent of a filter
  //    is to modify signal phase, such as differentiators and Hilbert
  //    filters.
  //
  //    % Example 1:
  //    %   Zero-phase filter a noisy ECG waveform using an IIR filter.
  //
  //    load noisysignals x;                    % noisy waveform
  //    [b,a] = butter(12,0.2,'low');           % IIR filter design
  //    y = filtfilt(b,a,x);                    % zero-phase filtering
  //    y2 = filter(b,a,x);                     % conventional filtering
  //    plot(x,'k-.'); grid on ; hold on
  //    plot([y y2],'LineWidth',1.5);
  //    legend('Noisy ECG','Zero-phase Filtering','Conventional Filtering');
  //
  //    % Example 2:
  //    %   Use the designfilt function to design a highpass IIR digital
  //    %   filter with order 4, passband frequency of 75 KHz, and a passband
  //    %   ripple of 0.2 dB. Sample rate is 200 KHz. Apply zero-phase
  //    %   filtering to a vector of data.
  //
  //    D = designfilt('highpassiir', 'FilterOrder', 4, ...
  //             'PassbandFrequency', 75e3, 'PassbandRipple', 0.2,...
  //             'SampleRate', 200e3);
  //
  //    x = rand(1000,1);
  //    y = filtfilt(D,x);
  //
  //    See also FILTER, SOSFILT.
  //    References:
  //      [1] Sanjit K. Mitra, Digital Signal Processing, 2nd ed.,
  //          McGraw-Hill, 2001
  //      [2] Fredrik Gustafsson, Determining the initial states in forward-
  //          backward filtering, IEEE Transactions on Signal Processing,
  //          pp. 988-992, April 1996, Volume 44, Issue 4
  //    Copyright 1988-2022 The MathWorks, Inc.
  //
  //  For filtfilt(D,X), the inputs passed to the function are B,A,X thanks
  //  to the method of D.
  //  Input validation
  // 'ec_filtfilt_fp:89' lfn = coder.target('MATLAB');
  // 'ec_filtfilt_fp:90' if ~lfn
  // 'ec_filtfilt_fp:91'
  // validateattributes(b,{'double','single'},{'2d','finite','nonempty'},'filtfilt');
  // 'ec_filtfilt_fp:92'
  // validateattributes(a,{'double','single'},{'2d','finite','vector','nonempty'},'filtfilt');
  // 'ec_filtfilt_fp:93'
  // validateattributes(x,{'double','single'},{'2d','finite','nonempty'},'filtfilt');
  //  Run
  // 'ec_filtfilt_fp:99' y = coder.nullcopy(x);
  yc5_outdatedOnCpu = false;
  i = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = cpu_x->size[0];
  cpu_y->size[1] = cpu_x->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(cpu_y, i, &emlrtRTEI);
  y_needsGpuEnsureCapacity = true;
  //  Loop across chans
  // 'ec_filtfilt_fp:102' for ch = 1:width(x)
  b_i = cpu_x->size[1];
  if (cpu_x->size[1] - 1 >= 0) {
    i1 = static_cast<int32_T>(L);
    if (static_cast<int32_T>(L) - 1 >= 0) {
      if (nf + 1.0F < 2.0F) {
        i2 = 1;
        i3 = 1;
        i4 = 0;
      } else {
        i2 = static_cast<int32_T>(nf + 1.0F);
        i3 = -1;
        i4 = 2;
      }
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      csz_idx_0 = div_s32(i4 - i2, i3) + 1;
    }
  }
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_fv, 1, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_xt, 1, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu__1, 1, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_zo, 1, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_yc2, 1, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_zo, 1, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_c, 1, &i_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_yc3, 1, &j_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu__2, 1, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_zo, 1, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_yc5, 1, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_b, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_a, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_z, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_b, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_a, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_b, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_a, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&e_cpu_b, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&e_cpu_a, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_yc3, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_z, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f_cpu_b, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f_cpu_a, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_yc2, 1, &d_emlrtRTEI, true);
  profileLoopStart("ec_filtfilt_fp_loop_0", __LINE__, (b_i - 1) + 1, "");
  for (int32_T ch{0}; ch < b_i; ch++) {
    int32_T i5;
    boolean_T validLaunchParams;
    // 'ec_filtfilt_fp:103' y(:,ch) = filtfilt_lfn(x(:,ch),b,a,z,nf,L);
    i = cpu_fv->size[0];
    cpu_fv->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_fv, i, &b_emlrtRTEI);
    i = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (x_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
    if (x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_filtfilt_fp_kernel1#" MW_AT_LINE);
      ec_filtfilt_fp_kernel1<<<grid, block>>>(ch, gpu_x, i, gpu_fv,
                                              cpu_x->size[0U]);
    }
    fv_outdatedOnGpu = false;
    //  oneCh_lfn %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //  Perform filtering of input data with no phase distortion
    //
    //  xc: matrix of input data
    //  yc: matrix of output data, same dimensions as xc
    //  a,b: IIR coefficients, both of same order/length
    //  zi: initial states
    //  nfact: scalar
    //  L: scalar
    //  odt: output data type
    //
    //  Same comments as in ffOneChan, except here we need to use bsxfun.
    //  Instead of doing scalar subtraction with a vector, we are doing
    //  vector addition with a matrix.  bsxfun replicates the vector
    //  for us.
    //
    //  We also take care to preserve column dimensions
    // coder.gpu.nokernel(); % No kernel due to dependencies across loops
    // coder.gpu.kernelfun; % Add kernelfun pragma to trigger kernel creation
    //  Loop by L
    // 'ec_filtfilt_fp:131' for ii = 1:L
    profileLoopStart("ec_filtfilt_fp_loop_1", __LINE__, (i1 - 1) + 1, "");
    for (int32_T ii{0}; ii < i1; ii++) {
      int32_T bcoef;
      int32_T i6;
      int32_T i7;
      real32_T a;
      boolean_T fv_outdatedOnCpu;
      // 'ec_filtfilt_fp:132' xt = bsxfun(@minus, 2*x(1,:),x(nf(1,1)+1:-1:2,:));
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
      fv_outdatedOnCpu = false;
      a = 2.0F * cpu_fv->data[0];
      xt_outdatedOnCpu = false;
      yc3_outdatedOnCpu = false;
      i = cpu_xt->size[0];
      cpu_xt->size[0] = csz_idx_0;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_xt, i, &c_emlrtRTEI);
      if (csz_idx_0 != 0) {
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(csz_idx_0 - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_xt, &gpu_xt, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_filtfilt_fp_kernel2#" MW_AT_LINE);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          ec_filtfilt_fp_kernel2<<<grid, block>>>(
              gpu_fv, static_cast<int32_T>(div_s32(i4 - i2, i3) + 1 != 1), i3,
              i2, a, csz_idx_0 - 1, gpu_xt);
        }
        xt_outdatedOnCpu = true;
      }
      // 'ec_filtfilt_fp:133' [~,zo] =
      // filter(b(:,ii),a(:,ii),xt,z(:,ii)*xt(1,:),1);
      if (xt_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_xt, &gpu_xt);
      }
      a = cpu_xt->data[0];
      i = b_cpu_b->size[0];
      b_cpu_b->size[0] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(b_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (zo_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_b, &gpu_b, !b_outdatedOnGpu);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(b_cpu_b, &b_gpu_b, true);
      if (b_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_b, cpu_b);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel3#" MW_AT_LINE);
        ec_filtfilt_fp_kernel3<<<grid, block>>>(gpu_b, i, b_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(b_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (yc3_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_a, &gpu_a, !a_outdatedOnGpu);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(b_cpu_a, &b_gpu_a, true);
      if (a_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_a, cpu_a);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel4#" MW_AT_LINE);
        ec_filtfilt_fp_kernel4<<<grid, block>>>(gpu_a, i, b_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      i = b_cpu_z->size[0];
      b_cpu_z->size[0] = cpu_z->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(b_cpu_z, i, &e_emlrtRTEI);
      i = cpu_z->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (z_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(b_cpu_z, &b_gpu_z, true);
      if (z_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_z, cpu_z);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel5#" MW_AT_LINE);
        ec_filtfilt_fp_kernel5<<<grid, block>>>(a, gpu_z, i, b_gpu_z);
      }
      z_outdatedOnGpu = false;
      yc2_outdatedOnCpu = true;
      nvtxMarkA("#filter#" MW_AT_LINE);
      coder::filter(b_cpu_b, &b_outdatedOnCpu, &b_gpu_b, &b_outdatedOnGpu,
                    b_cpu_a, &a_outdatedOnCpu, &b_gpu_a, &a_outdatedOnGpu,
                    cpu_xt, &gpu_xt, &yc3_outdatedOnCpu, b_cpu_z,
                    &yc2_outdatedOnCpu, &b_gpu_z, &z_outdatedOnGpu, cpu__1,
                    &yc5_outdatedOnCpu, &gpu__1, &yc5_outdatedOnGpu, cpu_zo,
                    &yc3_outdatedOnGpu, &gpu_zo, &zo_outdatedOnGpu);
      //  outer product
      // 'ec_filtfilt_fp:134' [yc2,zo] = filter(b(:,ii),a(:,ii),x,zo,1);
      i = c_cpu_b->size[0];
      c_cpu_b->size[0] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(c_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(c_cpu_b, &c_gpu_b, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel6#" MW_AT_LINE);
        ec_filtfilt_fp_kernel6<<<grid, block>>>(gpu_b, i, c_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = c_cpu_a->size[0];
      c_cpu_a->size[0] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(c_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(c_cpu_a, &c_gpu_a, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel7#" MW_AT_LINE);
        ec_filtfilt_fp_kernel7<<<grid, block>>>(gpu_a, i, c_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      nvtxMarkA("#filter#" MW_AT_LINE);
      coder::filter(c_cpu_b, &b_outdatedOnCpu, &c_gpu_b, &b_outdatedOnGpu,
                    c_cpu_a, &a_outdatedOnCpu, &c_gpu_a, &a_outdatedOnGpu,
                    cpu_fv, &gpu_fv, &fv_outdatedOnGpu, cpu_zo,
                    &yc3_outdatedOnGpu, &gpu_zo, &zo_outdatedOnGpu, cpu_yc2,
                    &yc2_outdatedOnCpu, &gpu_yc2, &yc2_outdatedOnGpu, b_cpu_zo,
                    &zo_outdatedOnCpu, &b_gpu_zo, &b_zo_outdatedOnGpu);
      // 'ec_filtfilt_fp:135' xt = bsxfun(@minus,
      // 2*x(end,:),x(end-1:-1:end-nf(1,1),:));
      a = static_cast<real32_T>(cpu_fv->size[0]) - nf;
      if (a >
          static_cast<real32_T>(static_cast<real_T>(cpu_fv->size[0]) - 1.0)) {
        i6 = 1;
        i7 = 1;
        i5 = 0;
      } else {
        i6 = static_cast<int32_T>(
            static_cast<real32_T>(static_cast<real_T>(cpu_fv->size[0]) - 1.0));
        i7 = -1;
        i5 = static_cast<int32_T>(a);
      }
      // 'ec_filtfilt_fp:136' yc3 = filter(b(:,ii),a(:,ii),xt,zo,1);
      if (fv_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
      }
      a = 2.0F * cpu_fv->data[cpu_fv->size[0] - 1];
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      i = div_s32(i5 - i6, i7) + 1;
      yc5_outdatedOnCpu = false;
      yc5_outdatedOnGpu = false;
      bcoef = cpu_c->size[0];
      cpu_c->size[0] = i;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_c, bcoef, &c_emlrtRTEI);
      if (i != 0) {
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, !fv_outdatedOnGpu);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_c, &gpu_c, true);
        if (fv_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_fv, cpu_fv);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_filtfilt_fp_kernel8#" MW_AT_LINE);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          ec_filtfilt_fp_kernel8<<<grid, block>>>(
              gpu_fv, static_cast<int32_T>(div_s32(i5 - i6, i7) + 1 != 1), i7,
              i6, a, i - 1, gpu_c);
        }
        yc5_outdatedOnCpu = true;
      }
      i = d_cpu_b->size[0];
      d_cpu_b->size[0] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(d_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(d_cpu_b, &d_gpu_b, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel9#" MW_AT_LINE);
        ec_filtfilt_fp_kernel9<<<grid, block>>>(gpu_b, i, d_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = d_cpu_a->size[0];
      d_cpu_a->size[0] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(d_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(d_cpu_a, &d_gpu_a, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel10#" MW_AT_LINE);
        ec_filtfilt_fp_kernel10<<<grid, block>>>(gpu_a, i, d_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      nvtxMarkA("#b_filter#" MW_AT_LINE);
      coder::b_filter(d_cpu_b, &b_outdatedOnCpu, &d_gpu_b, &b_outdatedOnGpu,
                      d_cpu_a, &a_outdatedOnCpu, &d_gpu_a, &a_outdatedOnGpu,
                      cpu_c, &gpu_c, &yc5_outdatedOnGpu, b_cpu_zo,
                      &zo_outdatedOnCpu, &b_gpu_zo, &b_zo_outdatedOnGpu,
                      cpu_yc3, &yc3_outdatedOnCpu, &gpu_yc3,
                      &yc3_outdatedOnGpu);
      // 'ec_filtfilt_fp:138' [~,zo] =
      // filter(b(:,ii),a(:,ii),yc3(end:-1:1,:),z(:,ii)*yc3(end,:),1);
      if (cpu_yc3->size[0] < 1) {
        i5 = 0;
        bcoef = 1;
        i6 = -1;
      } else {
        i5 = cpu_yc3->size[0] - 1;
        bcoef = -1;
        i6 = 0;
      }
      if (yc3_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_yc3, &gpu_yc3);
      }
      a = cpu_yc3->data[cpu_yc3->size[0] - 1];
      i = e_cpu_b->size[0];
      e_cpu_b->size[0] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(e_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(e_cpu_b, &e_gpu_b, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel11#" MW_AT_LINE);
        ec_filtfilt_fp_kernel11<<<grid, block>>>(gpu_b, i, e_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = e_cpu_a->size[0];
      e_cpu_a->size[0] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(e_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(e_cpu_a, &e_gpu_a, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel12#" MW_AT_LINE);
        ec_filtfilt_fp_kernel12<<<grid, block>>>(gpu_a, i, e_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      i = b_cpu_yc3->size[0];
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      b_cpu_yc3->size[0] = div_s32(i6 - i5, bcoef) + 1;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(b_cpu_yc3, i, &d_emlrtRTEI);
      i = (i6 - i5) / bcoef;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_yc3, &gpu_yc3, !yc3_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(b_cpu_yc3, &b_gpu_yc3, true);
      if (yc3_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_yc3, cpu_yc3);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel13#" MW_AT_LINE);
        ec_filtfilt_fp_kernel13<<<grid, block>>>(gpu_yc3, bcoef, i5, i,
                                                 b_gpu_yc3);
      }
      yc3_outdatedOnGpu = false;
      i = c_cpu_z->size[0];
      c_cpu_z->size[0] = cpu_z->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(c_cpu_z, i, &e_emlrtRTEI);
      i = cpu_z->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      z_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(c_cpu_z, &c_gpu_z, true);
      z_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel14#" MW_AT_LINE);
        ec_filtfilt_fp_kernel14<<<grid, block>>>(a, gpu_z, i, c_gpu_z);
      }
      xt_outdatedOnCpu = false;
      yc2_outdatedOnCpu = true;
      nvtxMarkA("#filter#" MW_AT_LINE);
      coder::filter(e_cpu_b, &b_outdatedOnCpu, &e_gpu_b, &b_outdatedOnGpu,
                    e_cpu_a, &a_outdatedOnCpu, &e_gpu_a, &a_outdatedOnGpu,
                    b_cpu_yc3, &b_gpu_yc3, &yc3_outdatedOnGpu, c_cpu_z,
                    &yc2_outdatedOnCpu, &c_gpu_z, &xt_outdatedOnCpu, cpu__2,
                    &yc5_outdatedOnCpu, &gpu__2, &yc5_outdatedOnGpu, c_cpu_zo,
                    &zo_outdatedOnCpu, &c_gpu_zo, &b_zo_outdatedOnGpu);
      //  outer product
      // 'ec_filtfilt_fp:139' yc5 =
      // filter(b(:,ii),a(:,ii),yc2(end:-1:1,:),zo,1);
      if (cpu_yc2->size[0] < 1) {
        i5 = 0;
        bcoef = 1;
        i6 = -1;
      } else {
        i5 = cpu_yc2->size[0] - 1;
        bcoef = -1;
        i6 = 0;
      }
      i = f_cpu_b->size[0];
      f_cpu_b->size[0] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(f_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      zo_outdatedOnGpu = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(f_cpu_b, &f_gpu_b, true);
      b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel15#" MW_AT_LINE);
        ec_filtfilt_fp_kernel15<<<grid, block>>>(gpu_b, i, f_gpu_b);
      }
      xt_outdatedOnCpu = false;
      b_outdatedOnCpu = true;
      i = f_cpu_a->size[0];
      f_cpu_a->size[0] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(f_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      yc3_outdatedOnGpu = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(f_cpu_a, &f_gpu_a, true);
      a_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel16#" MW_AT_LINE);
        ec_filtfilt_fp_kernel16<<<grid, block>>>(gpu_a, i, f_gpu_a);
      }
      yc3_outdatedOnCpu = false;
      a_outdatedOnCpu = true;
      i = b_cpu_yc2->size[0];
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      b_cpu_yc2->size[0] = div_s32(i6 - i5, bcoef) + 1;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(b_cpu_yc2, i, &d_emlrtRTEI);
      i = (i6 - i5) / bcoef;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_yc2, &gpu_yc2, !yc2_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(b_cpu_yc2, &b_gpu_yc2, true);
      if (yc2_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_yc2, cpu_yc2);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel17#" MW_AT_LINE);
        ec_filtfilt_fp_kernel17<<<grid, block>>>(gpu_yc2, bcoef, i5, i,
                                                 b_gpu_yc2);
      }
      yc2_outdatedOnGpu = false;
      nvtxMarkA("#b_filter#" MW_AT_LINE);
      coder::b_filter(f_cpu_b, &b_outdatedOnCpu, &f_gpu_b, &xt_outdatedOnCpu,
                      f_cpu_a, &a_outdatedOnCpu, &f_gpu_a, &yc3_outdatedOnCpu,
                      b_cpu_yc2, &b_gpu_yc2, &yc2_outdatedOnGpu, c_cpu_zo,
                      &zo_outdatedOnCpu, &c_gpu_zo, &b_zo_outdatedOnGpu,
                      cpu_yc5, &yc5_outdatedOnCpu, &gpu_yc5,
                      &yc5_outdatedOnGpu);
      // 'ec_filtfilt_fp:141' x = yc5(end:-1:1,:);
      if (cpu_yc5->size[0] < 1) {
        i5 = 0;
        bcoef = 1;
        i6 = -1;
      } else {
        i5 = cpu_yc5->size[0] - 1;
        bcoef = -1;
        i6 = 0;
      }
      i = cpu_fv->size[0];
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      cpu_fv->size[0] = div_s32(i6 - i5, bcoef) + 1;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_fv, i, &d_emlrtRTEI);
      i = (i6 - i5) / bcoef;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_yc5, &gpu_yc5, !yc5_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
      if (yc5_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_yc5, cpu_yc5);
      }
      yc5_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_filtfilt_fp_kernel18#" MW_AT_LINE);
        ec_filtfilt_fp_kernel18<<<grid, block>>>(gpu_yc5, bcoef, i5, i, gpu_fv);
      }
      fv_outdatedOnGpu = false;
    }
    profileLoopEnd();
    //  LocalWords:  x b a nf L
    //  LocalWords:  Lx th zi nfilt xc yc IIR nfact xt unreversed nullcopy Npts
    //  nb na yout LocalWords:  Nchans filtord designfilt noisysignals
    //  highpassiir Sanjit Mitra LocalWords:  nd Graw Fredrik Gustafsson
    i5 = cpu_y->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i5 - 1), &grid, &block,
                            2147483647U);
    if (y_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_y, &gpu_y, true);
    }
    y_needsGpuEnsureCapacity = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_filtfilt_fp_kernel19#" MW_AT_LINE);
      ec_filtfilt_fp_kernel19<<<grid, block>>>(gpu_fv, ch, i5 - 1, gpu_y,
                                               cpu_y->size[0U]);
    }
    yc5_outdatedOnCpu = true;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_yc2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f_cpu_a);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f_cpu_b);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_z);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_yc3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&e_cpu_a);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&e_cpu_b);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_a);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_b);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_a);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_b);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_z);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_a);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_b);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_yc5);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu__2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_yc3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_c);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_yc2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu__1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_xt);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_fv);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (yc5_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &gpu_y);
  }
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_b);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_z);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_fv);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_xt);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu__1);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_c);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu__2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_yc5);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_b);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_z);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_b);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_a);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_b);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_a);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&e_gpu_b);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&e_gpu_a);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_z);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&f_gpu_b);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&f_gpu_a);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_yc2);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_filtfilt_fp.cu)

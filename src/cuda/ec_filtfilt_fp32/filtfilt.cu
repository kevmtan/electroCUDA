#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filtfilt.cu
//
// Code generation for function 'filtfilt'
//

// Include files
#include "filtfilt.h"
#include "bsxfun.h"
#include "fillIn.h"
#include "filter.h"
#include "filtfilt_data.h"
#include "filtfilt_emxutil.h"
#include "filtfilt_mexutil.h"
#include "filtfilt_types.h"
#include "introsort.h"
#include "repmat.h"
#include "rt_nonfinite.h"
#include "sparse.h"
#include "warning.h"
#include "MWLocationStringifyNvtx3.h"
#include "cs.h"
#include "makeCXSparseMatrix.h"
#include "nvtx3/nvToolsExt.h"
#include "solve_from_lu.h"
#include "solve_from_qr.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtMCInfo d_emlrtMCI{
    53,        // lineNo
    19,        // colNo
    "flt2str", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/lib/+coder/+internal/"
    "flt2str.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    123,                                                        // lineNo
    10,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    166,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    181,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    420,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo
    e_emlrtRTEI{
        76,                  // lineNo
        13,                  // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo f_emlrtRTEI{
    184,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    150,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    179,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    423,                                                        // lineNo
    36,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    356,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    304,                                                        // lineNo
    16,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    424,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    357,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    424,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    357,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    307,                                                        // lineNo
    20,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo
    q_emlrtRTEI{
        88,                  // lineNo
        13,                  // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo r_emlrtRTEI{
    181,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    307,                                                        // lineNo
    28,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    425,                                                        // lineNo
    23,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    425,                                                        // lineNo
    31,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    358,                                                        // lineNo
    23,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    308,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    358,                                                        // lineNo
    31,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    308,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    358,                                                        // lineNo
    39,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo bb_emlrtRTEI{
    309,                                                        // lineNo
    20,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo cb_emlrtRTEI{
    426,                                                        // lineNo
    38,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo db_emlrtRTEI{
    309,                                                        // lineNo
    28,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo eb_emlrtRTEI{
    427,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fb_emlrtRTEI{
    427,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo gb_emlrtRTEI{
    309,                                                        // lineNo
    36,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo hb_emlrtRTEI{
    359,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ib_emlrtRTEI{
    429,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo jb_emlrtRTEI{
    360,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo kb_emlrtRTEI{
    429,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo lb_emlrtRTEI{
    360,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo mb_emlrtRTEI{
    429,                                                        // lineNo
    37,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo nb_emlrtRTEI{
    373,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ob_emlrtRTEI{
    430,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo pb_emlrtRTEI{
    373,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo qb_emlrtRTEI{
    430,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo rb_emlrtRTEI{
    430,                                                        // lineNo
    34,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo sb_emlrtRTEI{
    374,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo tb_emlrtRTEI{
    432,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ub_emlrtRTEI{
    374,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo vb_emlrtRTEI{
    1,                                                          // lineNo
    14,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo wb_emlrtRTEI{
    300,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo xb_emlrtRTEI{
    416,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo yb_emlrtRTEI{
    423,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ac_emlrtRTEI{
    304,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo bc_emlrtRTEI{
    427,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo cc_emlrtRTEI{
    360,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo dc_emlrtRTEI{
    430,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ec_emlrtRTEI{
    374,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fc_emlrtRTEI{
    426,                                                        // lineNo
    10,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo hc_emlrtRTEI{
    713,                                                        // lineNo
    40,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ic_emlrtRTEI{
    223,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo jc_emlrtRTEI{
    224,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo kc_emlrtRTEI{
    716,                                                        // lineNo
    40,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo lc_emlrtRTEI{
    54,                                                               // lineNo
    9,                                                                // colNo
    "div",                                                            // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/div.m" // pName
};

static emlrtRTEInfo mc_emlrtRTEI{
    231,                                                        // lineNo
    25,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo nc_emlrtRTEI{
    232,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo oc_emlrtRTEI{
    119,                                                          // lineNo
    28,                                                           // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

static emlrtRTEInfo pc_emlrtRTEI{
    238,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo qc_emlrtRTEI{
    245,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    246,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo sc_emlrtRTEI{
    250,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    261,                                                        // lineNo
    40,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    1647,     // lineNo
    27,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    125,      // lineNo
    44,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo wc_emlrtRTEI{
    1676,     // lineNo
    5,        // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo xc_emlrtRTEI{
    13,                                                                // lineNo
    1,                                                                 // colNo
    "sparse",                                                          // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/sparfun/sparse.m" // pName
};

static emlrtRTEInfo yc_emlrtRTEI{
    261,                                                        // lineNo
    56,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ad_emlrtRTEI{
    261,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo
    bd_emlrtRTEI{
        388,           // lineNo
        38,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo
    cd_emlrtRTEI{
        405,           // lineNo
        46,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo
    dd_emlrtRTEI{
        399,           // lineNo
        46,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo ed_emlrtRTEI{
    232,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    190,      // lineNo
    13,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo gd_emlrtRTEI{
    261,                                                        // lineNo
    19,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo hd_emlrtRTEI{
    119,      // lineNo
    13,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo id_emlrtRTEI{
    120,      // lineNo
    13,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo jd_emlrtRTEI{
    125,      // lineNo
    13,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo
    kd_emlrtRTEI{
        457,           // lineNo
        63,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo
    ld_emlrtRTEI{
        394,           // lineNo
        25,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo md_emlrtRTEI{
    30,                    // lineNo
    21,                    // colNo
    "applyScalarFunction", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
    "applyScalarFunction.m" // pName
};

static emlrtRTEInfo nd_emlrtRTEI{
    733,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo od_emlrtRTEI{
    738,                                                        // lineNo
    20,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo pd_emlrtRTEI{
    731,                                                        // lineNo
    15,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo be_emlrtRTEI{
    505,                                                        // lineNo
    12,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ce_emlrtRTEI{
    661,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo de_emlrtRTEI{
    672,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ee_emlrtRTEI{
    667,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fe_emlrtRTEI{
    656,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ge_emlrtRTEI{
    550,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo he_emlrtRTEI{
    673,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ie_emlrtRTEI{
    551,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo je_emlrtRTEI{
    657,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ke_emlrtRTEI{
    452,                                                        // lineNo
    8,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[14]);

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location);

static void binary_expand_op_1(emxArray_real32_T *in1, real_T in2, int32_T in3,
                               const emxArray_real32_T *in4, int32_T in5,
                               const emxArray_real32_T *in6);

namespace coder {
static void filtfiltParser(const emxArray_real32_T *ctf,
                           const emxArray_real32_T *varargin_1,
                           emxArray_real32_T *B, emxArray_real32_T *A,
                           real_T *numStage, real_T *P, real_T *Q,
                           boolean_T *isNumNonempty, boolean_T *isDenNonempty);

static void findEffectiveFilterLen(emxArray_real32_T *coef,
                                   real32_T effLen_data[],
                                   int32_T effLen_size[1]);

static void getCoeffsAndInitialConditions(
    const emxArray_real32_T *num, const emxArray_real32_T *den, real_T numStage,
    real_T P, real_T Q, emxArray_real32_T *B, emxArray_real32_T *A,
    emxArray_real32_T *zi, real32_T *Letr);

} // namespace coder
static int32_T div_s32(int32_T numerator, int32_T denominator);

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[14]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[14]);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[14])
{
  static const int32_T dims[2]{1, 14};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 14);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  const mxArray *m3;
  nvtxRangePushA("#fcn#b_sprintf#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  m3 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                             "sprintf", true, location);
  nvtxRangePop();
  return m3;
}

static void binary_expand_op_1(emxArray_real32_T *in1, real_T in2, int32_T in3,
                               const emxArray_real32_T *in4, int32_T in5,
                               const emxArray_real32_T *in6)
{
  emxArray_real32_T *b_in4;
  int32_T i;
  int32_T in6_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#binary_expand_op_1#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  in6_idx_0 = in6->size[0];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_in4, 2, &lc_emlrtRTEI, true);
  i = b_in4->size[0] * b_in4->size[1];
  if (in6_idx_0 == 1) {
    b_in4->size[0] = in4->size[0];
  } else {
    b_in4->size[0] = in6_idx_0;
  }
  b_in4->size[1] = in5 + 1;
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(b_in4, i, &lc_emlrtRTEI);
  stride_0_0 = (in4->size[0] != 1);
  stride_1_0 = (in6_idx_0 != 1);
  profileLoopStart("binary_expand_op_1_loop_0", __LINE__, in5 + 1, "");
  for (i = 0; i <= in5; i++) {
    int32_T b;
    if (in6_idx_0 == 1) {
      b = in4->size[0];
    } else {
      b = in6_idx_0;
    }
    profileLoopStart("binary_expand_op_1_loop_2", __LINE__, (b - 1) + 1, "");
    for (int32_T i1{0}; i1 < b; i1++) {
      b_in4->data[i1 + b_in4->size[0] * i] =
          in4->data[i1 * stride_0_0 + in4->size[0] * i] /
          in6->data[i1 * stride_1_0];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  in6_idx_0 = static_cast<int32_T>(in2);
  profileLoopStart("binary_expand_op_1_loop_1", __LINE__, (in3 - 1) + 1, "");
  for (i = 0; i < in3; i++) {
    profileLoopStart("binary_expand_op_1_loop_3", __LINE__, (in6_idx_0 - 1) + 1,
                     "");
    for (int32_T i1{0}; i1 < in6_idx_0; i1++) {
      in1->data[i1 + in1->size[0] * i] =
          b_in4->data[i1 + static_cast<int32_T>(in2) * i];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_in4);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

//
//
namespace coder {
static void filtfiltParser(const emxArray_real32_T *ctf,
                           const emxArray_real32_T *varargin_1,
                           emxArray_real32_T *B, emxArray_real32_T *A,
                           real_T *numStage, real_T *P, real_T *Q,
                           boolean_T *isNumNonempty, boolean_T *isDenNonempty)
{
  emxArray_real32_T *NUM;
  int32_T b_numStage;
  int32_T numSV;
  int32_T u0;
  int32_T u1;
  boolean_T guard1;
  boolean_T guard2;
  boolean_T isDenNonemptyVector;
  boolean_T isDenVector;
  boolean_T isNumVector;
  nvtxRangePushA("#fcn#filtfiltParser#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  *isNumNonempty = ((ctf->size[0] != 0) && (ctf->size[1] != 0));
  *isDenNonempty = ((varargin_1->size[0] != 0) && (varargin_1->size[1] != 0));
  isNumVector = ((ctf->size[0] == 1) || (ctf->size[1] == 1));
  isDenVector = ((varargin_1->size[0] == 1) || (varargin_1->size[1] == 1));
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&NUM, 2, &ke_emlrtRTEI, true);
  u0 = NUM->size[0] * NUM->size[1];
  NUM->size[0] = ctf->size[0];
  NUM->size[1] = ctf->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(NUM, u0, &be_emlrtRTEI);
  profileLoopStart("filtfiltParser_loop_0", __LINE__,
                   (ctf->size[0] * ctf->size[1] - 1) + 1, "");
  for (u0 = 0; u0 < ctf->size[0] * ctf->size[1]; u0++) {
    NUM->data[u0] = ctf->data[u0];
  }
  profileLoopEnd();
  b_numStage = ctf->size[0];
  if ((*isDenNonempty) && isDenVector) {
    isDenNonemptyVector = true;
  } else {
    isDenNonemptyVector = false;
  }
  if ((ctf->size[1] == 6) && (ctf->size[0] == 1) && isDenNonemptyVector) {
    if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
      numSV = 0;
    } else {
      u0 = varargin_1->size[0];
      numSV = varargin_1->size[1];
      if (u0 >= numSV) {
        numSV = u0;
      }
    }
    if (numSV <= 2) {
      if (ctf->data[3] == 1.0F) {
        nvtxMarkA("#warning#" MW_AT_LINE);
        internal::warning();
      } else {
        nvtxMarkA("#b_warning#" MW_AT_LINE);
        internal::b_warning();
      }
    }
  }
  guard1 = false;
  guard2 = false;
  if ((*isNumNonempty) && isDenNonemptyVector && (ctf->size[1] == 6)) {
    if (ctf->size[0] > 1) {
      guard2 = true;
    } else if (ctf->data[3] == 1.0F) {
      if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
        numSV = 0;
      } else {
        u0 = varargin_1->size[0];
        numSV = varargin_1->size[1];
        if (u0 >= numSV) {
          numSV = u0;
        }
      }
      if (numSV <= 2) {
        guard2 = true;
      } else {
        guard1 = true;
      }
    } else {
      guard1 = true;
    }
  } else {
    guard1 = true;
  }
  if (guard2) {
    if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
      numSV = 0;
    } else {
      u0 = varargin_1->size[0];
      numSV = varargin_1->size[1];
      if (u0 >= numSV) {
        numSV = u0;
      }
    }
    if (static_cast<uint32_T>(numSV) ==
        static_cast<uint32_T>(ctf->size[0]) + 1U) {
      NUM->data[ctf->size[0] - 1] =
          varargin_1->data[ctf->size[0]] * ctf->data[ctf->size[0] - 1];
      NUM->data[(ctf->size[0] + NUM->size[0]) - 1] =
          varargin_1->data[ctf->size[0]] *
          ctf->data[(ctf->size[0] + ctf->size[0]) - 1];
      NUM->data[(ctf->size[0] + NUM->size[0] * 2) - 1] =
          varargin_1->data[ctf->size[0]] *
          ctf->data[(ctf->size[0] + ctf->size[0] * 2) - 1];
      numSV--;
    }
    profileLoopStart("filtfiltParser_loop_1", __LINE__, (numSV - 1) + 1, "");
    for (u0 = 0; u0 < numSV; u0++) {
      NUM->data[u0] *= varargin_1->data[u0];
      NUM->data[u0 + NUM->size[0]] *= varargin_1->data[u0];
      NUM->data[u0 + NUM->size[0] * 2] *= varargin_1->data[u0];
    }
    profileLoopEnd();
    u0 = B->size[0] * B->size[1];
    B->size[0] = NUM->size[0];
    B->size[1] = 3;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(B, u0, &ge_emlrtRTEI);
    u0 = A->size[0] * A->size[1];
    A->size[0] = NUM->size[0];
    A->size[1] = 3;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(A, u0, &ie_emlrtRTEI);
    profileLoopStart("filtfiltParser_loop_7", __LINE__, 2 + 1, "");
    for (u0 = 0; u0 < 3; u0++) {
      profileLoopStart("filtfiltParser_loop_8", __LINE__,
                       (NUM->size[0] - 1) + 1, "");
      for (numSV = 0; numSV < NUM->size[0]; numSV++) {
        B->data[numSV + B->size[0] * u0] = NUM->data[numSV + NUM->size[0] * u0];
        A->data[numSV + A->size[0] * u0] =
            NUM->data[numSV + NUM->size[0] * (u0 + 3)];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    numSV = 3;
    u1 = 3;
  }
  if (guard1) {
    if (isNumVector && isDenVector) {
      if ((ctf->size[0] == 0) || (ctf->size[1] == 0)) {
        numSV = 0;
      } else {
        u0 = ctf->size[0];
        numSV = ctf->size[1];
        if (u0 >= numSV) {
          numSV = u0;
        }
      }
      if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
        u1 = 0;
      } else {
        u0 = varargin_1->size[0];
        u1 = varargin_1->size[1];
        if (u0 >= u1) {
          u1 = u0;
        }
      }
      b_numStage = 1;
      u0 = B->size[0] * B->size[1];
      B->size[0] = 1;
      B->size[1] = numSV;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(B, u0, &fe_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_5", __LINE__, (numSV - 1) + 1, "");
      for (u0 = 0; u0 < numSV; u0++) {
        B->data[u0] = ctf->data[u0];
      }
      profileLoopEnd();
      u0 = A->size[0] * A->size[1];
      A->size[0] = 1;
      A->size[1] = u1;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(A, u0, &je_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_9", __LINE__, (u1 - 1) + 1, "");
      for (u0 = 0; u0 < u1; u0++) {
        A->data[u0] = varargin_1->data[u0];
      }
      profileLoopEnd();
    } else if ((*isNumNonempty) && (!isNumVector) &&
               ((varargin_1->size[0] == 1) && (varargin_1->size[1] == 1))) {
      u0 = B->size[0] * B->size[1];
      B->size[0] = ctf->size[0];
      B->size[1] = ctf->size[1];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(B, u0, &ce_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_2", __LINE__,
                       (ctf->size[0] * ctf->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < ctf->size[0] * ctf->size[1]; u0++) {
        B->data[u0] = ctf->data[u0];
      }
      profileLoopEnd();
      numSV = ctf->size[1];
      b_numStage = ctf->size[0];
      u1 = 1;
      nvtxMarkA("#repmat#" MW_AT_LINE);
      repmat(varargin_1, static_cast<real_T>(ctf->size[0]), A);
    } else if ((*isDenNonempty) && (!isDenVector) &&
               ((ctf->size[0] == 1) && (ctf->size[1] == 1))) {
      u0 = A->size[0] * A->size[1];
      A->size[0] = varargin_1->size[0];
      A->size[1] = varargin_1->size[1];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(A, u0, &ee_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_4", __LINE__,
                       (varargin_1->size[0] * varargin_1->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < varargin_1->size[0] * varargin_1->size[1]; u0++) {
        A->data[u0] = varargin_1->data[u0];
      }
      profileLoopEnd();
      u1 = varargin_1->size[1];
      b_numStage = varargin_1->size[0];
      numSV = 1;
      nvtxMarkA("#repmat#" MW_AT_LINE);
      repmat(ctf, static_cast<real_T>(varargin_1->size[0]), B);
    } else {
      u0 = B->size[0] * B->size[1];
      B->size[0] = ctf->size[0];
      B->size[1] = ctf->size[1];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(B, u0, &de_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_3", __LINE__,
                       (ctf->size[0] * ctf->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < ctf->size[0] * ctf->size[1]; u0++) {
        B->data[u0] = ctf->data[u0];
      }
      profileLoopEnd();
      u0 = A->size[0] * A->size[1];
      A->size[0] = varargin_1->size[0];
      A->size[1] = varargin_1->size[1];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(A, u0, &he_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_6", __LINE__,
                       (varargin_1->size[0] * varargin_1->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < varargin_1->size[0] * varargin_1->size[1]; u0++) {
        A->data[u0] = varargin_1->data[u0];
      }
      profileLoopEnd();
      numSV = ctf->size[1];
      b_numStage = ctf->size[0];
      u1 = varargin_1->size[1];
    }
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&NUM);
  *numStage = b_numStage;
  *P = numSV;
  *Q = u1;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

//
//
static void findEffectiveFilterLen(emxArray_real32_T *coef,
                                   real32_T effLen_data[],
                                   int32_T effLen_size[1])
{
  emxArray_boolean_T *x;
  emxArray_real32_T *varargin_1;
  int32_T ii_data[1];
  int32_T idx;
  int32_T k;
  int32_T nx;
  real32_T maxCoef;
  boolean_T exitg1;
  nvtxRangePushA("#fcn#findEffectiveFilterLen#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nx = coef->size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&varargin_1, 2, &pd_emlrtRTEI, true);
  idx = varargin_1->size[0] * varargin_1->size[1];
  varargin_1->size[0] = 1;
  varargin_1->size[1] = coef->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(varargin_1, idx, &md_emlrtRTEI);
  profileLoopStart("findEffectiveFilterLen_loop_0", __LINE__, (nx - 1) + 1, "");
  for (k = 0; k < nx; k++) {
    varargin_1->data[k] = std::abs(coef->data[k]);
  }
  profileLoopEnd();
  nx = varargin_1->size[1];
  maxCoef = varargin_1->data[0];
  profileLoopStart("findEffectiveFilterLen_loop_1", __LINE__, (nx - 2) + 1, "");
  for (idx = 0; idx <= nx - 2; idx++) {
    real32_T f;
    boolean_T p;
    f = varargin_1->data[idx + 1];
    if (std::isnan(f)) {
      p = false;
    } else if (std::isnan(maxCoef)) {
      p = true;
    } else {
      p = (maxCoef < f);
    }
    if (p) {
      maxCoef = f;
    }
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&varargin_1);
  if (maxCoef != 0.0F) {
    idx = coef->size[1];
    nx = coef->size[0] * coef->size[1];
    coef->size[0] = 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(coef, nx, &nd_emlrtRTEI);
    profileLoopStart("findEffectiveFilterLen_loop_2", __LINE__, (idx - 1) + 1,
                     "");
    for (nx = 0; nx < idx; nx++) {
      coef->data[nx] /= maxCoef;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxInit_boolean_T#" MW_AT_LINE);
  emxInit_boolean_T(&x, 1, &od_emlrtRTEI, true);
  idx = x->size[0];
  x->size[0] = coef->size[1];
  nvtxMarkA("#emxEnsureCapacity_boolean_T#" MW_AT_LINE);
  emxEnsureCapacity_boolean_T(x, idx, &od_emlrtRTEI);
  profileLoopStart("findEffectiveFilterLen_loop_3", __LINE__,
                   (coef->size[1] - 1) + 1, "");
  for (idx = 0; idx < coef->size[1]; idx++) {
    x->data[idx] = (coef->data[idx] != 0.0F);
  }
  profileLoopEnd();
  k = (x->size[0] >= 1);
  nx = x->size[0];
  idx = 0;
  exitg1 = false;
  nvtxRangePushA("#loop#findEffectiveFilterLen_whileloop_0##" MW_AT_LINE);
  while ((!exitg1) && (nx > 0)) {
    if (x->data[nx - 1]) {
      idx = 1;
      ii_data[0] = nx;
      exitg1 = true;
    } else {
      nx--;
    }
  }
  nvtxRangePop();
  nvtxMarkA("#emxFree_boolean_T#" MW_AT_LINE);
  emxFree_boolean_T(&x);
  if (k == 1) {
    if (idx == 0) {
      k = 0;
    }
  } else {
    k = (idx >= 1);
  }
  effLen_size[0] = k;
  profileLoopStart("findEffectiveFilterLen_loop_4", __LINE__, (k - 1) + 1, "");
  for (idx = 0; idx < k; idx++) {
    effLen_data[0] = static_cast<real32_T>(ii_data[0]);
  }
  profileLoopEnd();
  if (k == 0) {
    effLen_size[0] = 1;
    effLen_data[0] = 0.0F;
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

//
//
static void getCoeffsAndInitialConditions(const emxArray_real32_T *num,
                                          const emxArray_real32_T *den,
                                          real_T numStage, real_T P, real_T Q,
                                          emxArray_real32_T *B,
                                          emxArray_real32_T *A,
                                          emxArray_real32_T *zi, real32_T *Letr)
{
  static const int32_T iv[2]{1, 6};
  static const char_T rfmt[6]{'%', '1', '4', '.', '6', 'e'};
  emxArray_int32_T *b_in_colidx;
  emxArray_int32_T *b_in_rowidx;
  emxArray_int32_T *b_t;
  emxArray_int32_T *c_in_colidx;
  emxArray_int32_T *c_in_rowidx;
  emxArray_int32_T *cidxInt;
  emxArray_int32_T *in_colidx;
  emxArray_int32_T *in_rowidx;
  emxArray_int32_T *ridxInt;
  emxArray_int32_T *sortedIndices;
  emxArray_int32_T *t;
  emxArray_int32_T *y_colidx;
  emxArray_int32_T *y_rowidx;
  emxArray_real32_T *IDX;
  emxArray_real32_T *b_den;
  emxArray_real32_T *b_num;
  emxArray_real32_T *c_den;
  emxArray_real32_T *c_num;
  emxArray_real32_T *colIdx;
  emxArray_real32_T *rhs;
  emxArray_real32_T *rowIdx;
  emxArray_real_T *b;
  emxArray_real_T *b_in_d;
  emxArray_real_T *b_outBuff;
  emxArray_real_T *c_in_d;
  emxArray_real_T *in_d;
  emxArray_real_T *outBuff;
  emxArray_real_T *y_d;
  emxArray_real_T *yf;
  emxArray_real_T *zik;
  const mxArray *b_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *y;
  int32_T b_i;
  int32_T b_loop_ub;
  int32_T b_ns;
  int32_T c_ns;
  int32_T i;
  int32_T i10;
  int32_T i11;
  int32_T k;
  int32_T loop_ub;
  int32_T n;
  int32_T nc;
  int32_T numalloc{0};
  int32_T thism;
  real32_T b_tmp_data[1];
  real32_T tmp_data[1];
  real32_T M;
  real32_T ord;
  nvtxRangePushA("#fcn#getCoeffsAndInitialConditions#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  M = std::fmax(static_cast<real32_T>(P), static_cast<real32_T>(Q));
  ord = 0.0F;
  i = static_cast<int32_T>(numStage);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_num, 2, &hc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_den, 2, &kc_emlrtRTEI, true);
  profileLoopStart("getCoeffsAndInitialConditions_loop_0", __LINE__,
                   (i - 1) + 1, "");
  for (int32_T ns{0}; ns < i; ns++) {
    b_i = b_num->size[0] * b_num->size[1];
    b_num->size[0] = 1;
    b_num->size[1] = num->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(b_num, b_i, &hc_emlrtRTEI);
    profileLoopStart("getCoeffsAndInitialConditions_loop_1", __LINE__,
                     (num->size[1] - 1) + 1, "");
    for (b_i = 0; b_i < num->size[1]; b_i++) {
      b_num->data[b_i] = num->data[ns + num->size[0] * b_i];
    }
    profileLoopEnd();
    b_i = b_den->size[0] * b_den->size[1];
    b_den->size[0] = 1;
    b_den->size[1] = den->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(b_den, b_i, &kc_emlrtRTEI);
    profileLoopStart("getCoeffsAndInitialConditions_loop_3", __LINE__,
                     (den->size[1] - 1) + 1, "");
    for (b_i = 0; b_i < den->size[1]; b_i++) {
      b_den->data[b_i] = den->data[ns + den->size[0] * b_i];
    }
    int32_T tmp_size[1];
    profileLoopEnd();
    nvtxMarkA("#findEffectiveFilterLen#" MW_AT_LINE);
    findEffectiveFilterLen(b_num, tmp_data, tmp_size);
    nvtxMarkA("#findEffectiveFilterLen#" MW_AT_LINE);
    findEffectiveFilterLen(b_den, b_tmp_data, tmp_size);
    ord = (ord + std::fmax(tmp_data[0], b_tmp_data[0])) - 1.0F;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_den);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_num);
  b_i = B->size[0] * B->size[1];
  B->size[0] = static_cast<int32_T>(numStage);
  B->size[1] = static_cast<int32_T>(M);
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(B, b_i, &ic_emlrtRTEI);
  b_i = A->size[0] * A->size[1];
  A->size[0] = static_cast<int32_T>(numStage);
  A->size[1] = static_cast<int32_T>(M);
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(A, b_i, &jc_emlrtRTEI);
  profileLoopStart(
      "getCoeffsAndInitialConditions_loop_2", __LINE__,
      (static_cast<int32_T>(numStage) * static_cast<int32_T>(M) - 1) + 1, "");
  for (b_i = 0; b_i < static_cast<int32_T>(numStage) * static_cast<int32_T>(M);
       b_i++) {
    B->data[b_i] = 0.0F;
    A->data[b_i] = 0.0F;
  }
  profileLoopEnd();
  if (P < 1.0) {
    n = 0;
    i = 0;
  } else {
    n = static_cast<int32_T>(P);
    i = static_cast<int32_T>(P);
  }
  if (num->size[0] == den->size[0]) {
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&c_num, 2, &lc_emlrtRTEI, true);
    b_i = c_num->size[0] * c_num->size[1];
    c_num->size[0] = num->size[0];
    c_num->size[1] = n;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(c_num, b_i, &lc_emlrtRTEI);
    profileLoopStart("getCoeffsAndInitialConditions_loop_4", __LINE__,
                     (n - 1) + 1, "");
    for (b_i = 0; b_i < n; b_i++) {
      profileLoopStart("getCoeffsAndInitialConditions_loop_5", __LINE__,
                       (num->size[0] - 1) + 1, "");
      for (k = 0; k < num->size[0]; k++) {
        c_num->data[k + c_num->size[0] * b_i] =
            num->data[k + num->size[0] * b_i] / den->data[k];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    thism = static_cast<int32_T>(numStage);
    profileLoopStart("getCoeffsAndInitialConditions_loop_6", __LINE__,
                     (i - 1) + 1, "");
    for (b_i = 0; b_i < i; b_i++) {
      profileLoopStart("getCoeffsAndInitialConditions_loop_8", __LINE__,
                       (thism - 1) + 1, "");
      for (k = 0; k < thism; k++) {
        B->data[k + B->size[0] * b_i] =
            c_num->data[k + static_cast<int32_T>(numStage) * b_i];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&c_num);
  } else {
    nvtxMarkA("#binary_expand_op_1#" MW_AT_LINE);
    binary_expand_op_1(B, numStage, i, num, n - 1, den);
  }
  if (Q < 1.0) {
    n = 0;
  } else {
    n = static_cast<int32_T>(Q);
  }
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_den, 2, &lc_emlrtRTEI, true);
  b_i = c_den->size[0] * c_den->size[1];
  c_den->size[0] = den->size[0];
  c_den->size[1] = n;
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(c_den, b_i, &lc_emlrtRTEI);
  profileLoopStart("getCoeffsAndInitialConditions_loop_7", __LINE__,
                   (n - 1) + 1, "");
  for (b_i = 0; b_i < n; b_i++) {
    profileLoopStart("getCoeffsAndInitialConditions_loop_9", __LINE__,
                     (den->size[0] - 1) + 1, "");
    for (k = 0; k < den->size[0]; k++) {
      c_den->data[k + c_den->size[0] * b_i] =
          den->data[k + den->size[0] * b_i] / den->data[k];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  thism = static_cast<int32_T>(numStage);
  if (Q < 1.0) {
    n = 0;
  } else {
    n = static_cast<int32_T>(Q);
  }
  profileLoopStart("getCoeffsAndInitialConditions_loop_10", __LINE__,
                   (n - 1) + 1, "");
  for (b_i = 0; b_i < n; b_i++) {
    profileLoopStart("getCoeffsAndInitialConditions_loop_11", __LINE__,
                     (thism - 1) + 1, "");
    for (k = 0; k < thism; k++) {
      A->data[k + A->size[0] * b_i] =
          c_den->data[k + static_cast<int32_T>(numStage) * b_i];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_den);
  if (M > 1.0F) {
    real_T tol;
    int32_T i7;
    int32_T i9;
    b_i = zi->size[0] * zi->size[1];
    zi->size[0] = static_cast<int32_T>(M - 1.0F);
    zi->size[1] = static_cast<int32_T>(numStage);
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(zi, b_i, &mc_emlrtRTEI);
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&rhs, 1, &ed_emlrtRTEI, true);
    b_i = rhs->size[0];
    rhs->size[0] = static_cast<int32_T>(M - 1.0F);
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(rhs, b_i, &nc_emlrtRTEI);
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&IDX, 2, &pc_emlrtRTEI, true);
    if (M - 1.0F >= 1.07374182E+9F) {
      tol = M - 1.0F;
      b_i = IDX->size[0] * IDX->size[1];
      IDX->size[0] = 1;
      IDX->size[1] = static_cast<int32_T>((M - 1.0F) - 1.0) + 1;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(IDX, b_i, &pc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_13", __LINE__,
                       static_cast<int32_T>(tol - 1.0) + 1, "");
      for (b_i = 0; b_i <= static_cast<int32_T>(tol - 1.0); b_i++) {
        IDX->data[b_i] = static_cast<real32_T>(static_cast<real_T>(b_i) + 1.0);
      }
      profileLoopEnd();
    } else {
      n = static_cast<int32_T>(std::floor(M - 1.0F));
      b_i = IDX->size[0] * IDX->size[1];
      IDX->size[0] = 1;
      IDX->size[1] = static_cast<int32_T>(std::floor(M - 1.0F));
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(IDX, b_i, &oc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_12", __LINE__,
                       (n - 1) + 1, "");
      for (k = 0; k < n; k++) {
        IDX->data[k] = static_cast<real32_T>(k + 1);
      }
      profileLoopEnd();
    }
    if (IDX->size[1] < 2) {
      n = 0;
      i = 0;
    } else {
      n = 1;
      i = IDX->size[1];
    }
    if (static_cast<real_T>(static_cast<int32_T>(M - 1.0F)) - 1.0 < 1.0) {
      i7 = 1;
    } else {
      i7 = static_cast<int32_T>(M - 1.0F);
    }
    if (IDX->size[1] - 1 < 1) {
      thism = 0;
    } else {
      thism = IDX->size[1] - 1;
    }
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&rowIdx, 2, &qc_emlrtRTEI, true);
    b_i = rowIdx->size[0] * rowIdx->size[1];
    rowIdx->size[0] = 1;
    rowIdx->size[1] = ((IDX->size[1] + i) - n) + thism;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(rowIdx, b_i, &qc_emlrtRTEI);
    k = IDX->size[1];
    profileLoopStart("getCoeffsAndInitialConditions_loop_14", __LINE__,
                     (k - 1) + 1, "");
    for (b_i = 0; b_i < k; b_i++) {
      rowIdx->data[b_i] = IDX->data[b_i];
    }
    profileLoopEnd();
    k = i - n;
    profileLoopStart("getCoeffsAndInitialConditions_loop_15", __LINE__,
                     (k - 1) + 1, "");
    for (b_i = 0; b_i < k; b_i++) {
      rowIdx->data[b_i + IDX->size[1]] = IDX->data[n + b_i];
    }
    profileLoopEnd();
    profileLoopStart("getCoeffsAndInitialConditions_loop_16", __LINE__,
                     (thism - 1) + 1, "");
    for (b_i = 0; b_i < thism; b_i++) {
      rowIdx->data[((b_i + IDX->size[1]) + i) - n] = IDX->data[b_i];
    }
    profileLoopEnd();
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&colIdx, 2, &rc_emlrtRTEI, true);
    b_i = colIdx->size[0] * colIdx->size[1];
    colIdx->size[0] = 1;
    colIdx->size[1] = (((static_cast<int32_T>(M - 1.0F) + i) - n) + i) - n;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(colIdx, b_i, &rc_emlrtRTEI);
    k = static_cast<int32_T>(M - 1.0F);
    profileLoopStart("getCoeffsAndInitialConditions_loop_17", __LINE__,
                     (k - 1) + 1, "");
    for (b_i = 0; b_i < k; b_i++) {
      colIdx->data[b_i] = 1.0F;
    }
    profileLoopEnd();
    k = i - n;
    profileLoopStart("getCoeffsAndInitialConditions_loop_18", __LINE__,
                     (k - 1) + 1, "");
    for (b_i = 0; b_i < k; b_i++) {
      colIdx->data[b_i + static_cast<int32_T>(M - 1.0F)] = IDX->data[n + b_i];
    }
    profileLoopEnd();
    k = i - n;
    profileLoopStart("getCoeffsAndInitialConditions_loop_19", __LINE__,
                     (k - 1) + 1, "");
    for (b_i = 0; b_i < k; b_i++) {
      colIdx->data[((b_i + static_cast<int32_T>(M - 1.0F)) + i) - n] =
          IDX->data[n + b_i];
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&IDX);
    i9 = static_cast<int32_T>(numStage);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&y_d, 1, &gd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&y_colidx, 1, &gd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&y_rowidx, 1, &gd_emlrtRTEI, true);
    if (static_cast<int32_T>(numStage) - 1 >= 0) {
      if (M < 3.0F) {
        i10 = 0;
        i11 = 0;
      } else {
        i10 = 2;
        i11 = static_cast<int32_T>(M);
      }
      loop_ub = i11 - i10;
      b_loop_ub = i7;
      nc = colIdx->size[1];
      b_ns = rowIdx->size[1];
      c_ns = colIdx->size[1];
      numalloc = colIdx->size[1];
      if (numalloc < 1) {
        numalloc = 1;
      }
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&yf, 2, &fd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&ridxInt, 1, &hd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&cidxInt, 1, &id_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&sortedIndices, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&t, 1, &wc_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_t, 1, &wc_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b, 1, &yc_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&zik, 1, &ad_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&in_d, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&in_colidx, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&in_rowidx, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_in_d, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_in_colidx, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_in_rowidx, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_in_d, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&c_in_colidx, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&c_in_rowidx, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&outBuff, 1, &ld_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_outBuff, 1, &ld_emlrtRTEI, true);
    profileLoopStart("getCoeffsAndInitialConditions_loop_20", __LINE__,
                     (i9 - 1) + 1, "");
    for (int32_T ns{0}; ns < i9; ns++) {
      M = B->data[ns];
      n = rhs->size[0];
      b_i = rhs->size[0];
      rhs->size[0] = n;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(rhs, b_i, &sc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_21", __LINE__,
                       (n - 1) + 1, "");
      for (b_i = 0; b_i < n; b_i++) {
        rhs->data[b_i] = B->data[ns + B->size[0] * (b_i + 1)] -
                         M * A->data[ns + A->size[0] * (b_i + 1)];
      }
      profileLoopEnd();
      b_i = yf->size[0] * yf->size[1];
      yf->size[0] = 1;
      yf->size[1] = (((i11 - i10) + i7) + i7) - 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(yf, b_i, &tc_emlrtRTEI);
      yf->data[0] = A->data[ns + A->size[0]] + 1.0F;
      profileLoopStart("getCoeffsAndInitialConditions_loop_22", __LINE__,
                       (loop_ub - 1) + 1, "");
      for (b_i = 0; b_i < loop_ub; b_i++) {
        yf->data[b_i + 1] = A->data[ns + A->size[0] * (i10 + b_i)];
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_23", __LINE__,
                       (b_loop_ub - 2) + 1, "");
      for (b_i = 0; b_i <= b_loop_ub - 2; b_i++) {
        yf->data[((b_i + i11) - i10) + 1] = 1.0;
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_24", __LINE__,
                       (b_loop_ub - 2) + 1, "");
      for (b_i = 0; b_i <= b_loop_ub - 2; b_i++) {
        yf->data[((b_i + i11) - i10) + i7] = -1.0;
      }
      profileLoopEnd();
      b_i = ridxInt->size[0];
      ridxInt->size[0] = rowIdx->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(ridxInt, b_i, &uc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_25", __LINE__,
                       (b_ns - 1) + 1, "");
      for (k = 0; k < b_ns; k++) {
        ridxInt->data[k] = static_cast<int32_T>(rowIdx->data[k]);
      }
      profileLoopEnd();
      b_i = cidxInt->size[0];
      cidxInt->size[0] = colIdx->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cidxInt, b_i, &uc_emlrtRTEI);
      b_i = sortedIndices->size[0];
      sortedIndices->size[0] = colIdx->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(sortedIndices, b_i, &vc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_26", __LINE__,
                       (c_ns - 1) + 1, "");
      for (k = 0; k < c_ns; k++) {
        cidxInt->data[k] = static_cast<int32_T>(colIdx->data[k]);
        sortedIndices->data[k] = k + 1;
      }
      profileLoopEnd();
      nvtxMarkA("#introsort#" MW_AT_LINE);
      internal::introsort(sortedIndices, cidxInt->size[0], cidxInt, ridxInt);
      n = cidxInt->size[0];
      b_i = t->size[0];
      t->size[0] = cidxInt->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(t, b_i, &wc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_27", __LINE__,
                       (cidxInt->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < cidxInt->size[0]; b_i++) {
        t->data[b_i] = cidxInt->data[b_i];
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_28", __LINE__,
                       (n - 1) + 1, "");
      for (k = 0; k < n; k++) {
        cidxInt->data[k] = t->data[sortedIndices->data[k] - 1];
      }
      profileLoopEnd();
      n = ridxInt->size[0];
      b_i = b_t->size[0];
      b_t->size[0] = ridxInt->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(b_t, b_i, &wc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_29", __LINE__,
                       (ridxInt->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < ridxInt->size[0]; b_i++) {
        b_t->data[b_i] = ridxInt->data[b_i];
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_30", __LINE__,
                       (n - 1) + 1, "");
      for (k = 0; k < n; k++) {
        ridxInt->data[k] = b_t->data[sortedIndices->data[k] - 1];
      }
      profileLoopEnd();
      n = ridxInt->size[0];
      thism = ridxInt->data[0];
      profileLoopStart("getCoeffsAndInitialConditions_loop_31", __LINE__,
                       (n - 2) + 1, "");
      for (i = 0; i <= n - 2; i++) {
        if (thism < ridxInt->data[i + 1]) {
          thism = ridxInt->data[i + 1];
        }
      }
      profileLoopEnd();
      i = cidxInt->data[cidxInt->size[0] - 1];
      b_i = y_d->size[0];
      y_d->size[0] = numalloc;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(y_d, b_i, &xc_emlrtRTEI);
      if (numalloc - 1 >= 0) {
        std::memset(&y_d->data[0], 0,
                    static_cast<uint32_T>(numalloc) * sizeof(real_T));
      }
      b_i = y_colidx->size[0];
      y_colidx->size[0] = cidxInt->data[cidxInt->size[0] - 1] + 1;
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(y_colidx, b_i, &xc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_32", __LINE__,
                       cidxInt->data[cidxInt->size[0] - 1] + 1, "");
      for (b_i = 0; b_i <= cidxInt->data[cidxInt->size[0] - 1]; b_i++) {
        y_colidx->data[b_i] = 0;
      }
      profileLoopEnd();
      y_colidx->data[0] = 1;
      b_i = y_rowidx->size[0];
      y_rowidx->size[0] = numalloc;
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(y_rowidx, b_i, &xc_emlrtRTEI);
      if (numalloc - 1 >= 0) {
        std::memset(&y_rowidx->data[0], 0,
                    static_cast<uint32_T>(numalloc) * sizeof(int32_T));
      }
      n = 0;
      profileLoopStart("getCoeffsAndInitialConditions_loop_33", __LINE__,
                       (i - 1) + 1, "");
      for (k = 0; k < i; k++) {
        nvtxRangePushA(
            "#loop#getCoeffsAndInitialConditions_whileloop_0##" MW_AT_LINE);
        while ((n + 1 <= nc) && (cidxInt->data[n] == k + 1)) {
          y_rowidx->data[n] = ridxInt->data[n];
          n++;
        }
        nvtxRangePop();
        y_colidx->data[k + 1] = n + 1;
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_34", __LINE__,
                       (nc - 1) + 1, "");
      for (k = 0; k < nc; k++) {
        y_d->data[k] = yf->data[sortedIndices->data[k] - 1];
      }
      profileLoopEnd();
      i = cidxInt->data[cidxInt->size[0] - 1];
      nvtxMarkA("#sparse_fillIn#" MW_AT_LINE);
      sparse_fillIn(y_d, y_colidx, y_rowidx);
      b_i = b->size[0];
      b->size[0] = rhs->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b, b_i, &yc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_35", __LINE__,
                       (rhs->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < rhs->size[0]; b_i++) {
        b->data[b_i] = rhs->data[b_i];
      }
      profileLoopEnd();
      if ((thism == 0) || (cidxInt->data[cidxInt->size[0] - 1] == 0)) {
        b_i = zik->size[0];
        zik->size[0] = cidxInt->data[cidxInt->size[0] - 1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(zik, b_i, &ad_emlrtRTEI);
        if (i - 1 >= 0) {
          std::memset(&zik->data[0], 0,
                      static_cast<uint32_T>(i) * sizeof(real_T));
        }
      } else if (b->size[0] == cidxInt->data[cidxInt->size[0] - 1]) {
        cs_di *b_cxA;
        cs_din *b_N;
        cs_dis *b_S;
        if (thism < cidxInt->data[cidxInt->size[0] - 1]) {
          nvtxMarkA("#sparse_ctranspose#" MW_AT_LINE);
          sparse_ctranspose(y_d, y_colidx, y_rowidx, thism,
                            cidxInt->data[cidxInt->size[0] - 1], b_in_d,
                            b_in_colidx, b_in_rowidx, &n, &i);
          b_cxA = makeCXSparseMatrix(
              b_in_colidx->data[b_in_colidx->size[0] - 1] - 1, i, n,
              &b_in_colidx->data[0], &b_in_rowidx->data[0], &b_in_d->data[0]);
        } else {
          b_cxA = makeCXSparseMatrix(y_colidx->data[y_colidx->size[0] - 1] - 1,
                                     cidxInt->data[cidxInt->size[0] - 1], thism,
                                     &y_colidx->data[0], &y_rowidx->data[0],
                                     &y_d->data[0]);
        }
        b_S = cs_di_sqr(2, b_cxA, 0);
        b_N = cs_di_lu(b_cxA, b_S, 1);
        cs_di_spfree(b_cxA);
        if (b_N == nullptr) {
          cs_di *c_cxA;
          cs_din *c_N;
          cs_dis *c_S;
          nvtxMarkA("#c_warning#" MW_AT_LINE);
          internal::c_warning();
          cs_di_sfree(b_S);
          cs_di_nfree(b_N);
          if (thism < cidxInt->data[cidxInt->size[0] - 1]) {
            nvtxMarkA("#sparse_ctranspose#" MW_AT_LINE);
            sparse_ctranspose(y_d, y_colidx, y_rowidx, thism,
                              cidxInt->data[cidxInt->size[0] - 1], c_in_d,
                              c_in_colidx, c_in_rowidx, &n, &i);
            c_cxA = makeCXSparseMatrix(
                c_in_colidx->data[c_in_colidx->size[0] - 1] - 1, i, n,
                &c_in_colidx->data[0], &c_in_rowidx->data[0], &c_in_d->data[0]);
          } else {
            c_cxA = makeCXSparseMatrix(
                y_colidx->data[y_colidx->size[0] - 1] - 1,
                cidxInt->data[cidxInt->size[0] - 1], thism, &y_colidx->data[0],
                &y_rowidx->data[0], &y_d->data[0]);
          }
          c_S = cs_di_sqr(2, c_cxA, 1);
          c_N = cs_di_qr(c_cxA, c_S);
          cs_di_spfree(c_cxA);
          qr_rank_di(c_N, &tol);
          b_i = zik->size[0];
          zik->size[0] = cidxInt->data[cidxInt->size[0] - 1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(zik, b_i, &bd_emlrtRTEI);
          if (b->size[0] < cidxInt->data[cidxInt->size[0] - 1]) {
            b_i = b_outBuff->size[0];
            b_outBuff->size[0] = cidxInt->data[cidxInt->size[0] - 1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(b_outBuff, b_i, &dd_emlrtRTEI);
          } else {
            b_i = b_outBuff->size[0];
            b_outBuff->size[0] = b->size[0];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(b_outBuff, b_i, &cd_emlrtRTEI);
          }
          k = b->size[0];
          profileLoopStart("getCoeffsAndInitialConditions_loop_40", __LINE__,
                           (k - 1) + 1, "");
          for (b_i = 0; b_i < k; b_i++) {
            b_outBuff->data[b_i] = b->data[b_i];
          }
          profileLoopEnd();
          solve_from_qr_di(c_N, c_S, (double *)&b_outBuff->data[0], b->size[0],
                           cidxInt->data[cidxInt->size[0] - 1]);
          n = cidxInt->data[cidxInt->size[0] - 1];
          profileLoopStart("getCoeffsAndInitialConditions_loop_41", __LINE__,
                           (n - 1) + 1, "");
          for (b_i = 0; b_i < n; b_i++) {
            zik->data[b_i] = b_outBuff->data[b_i];
          }
          profileLoopEnd();
          cs_di_sfree(c_S);
          cs_di_nfree(c_N);
        } else {
          b_i = zik->size[0];
          zik->size[0] = b->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(zik, b_i, &ad_emlrtRTEI);
          profileLoopStart("getCoeffsAndInitialConditions_loop_37", __LINE__,
                           (b->size[0] - 1) + 1, "");
          for (b_i = 0; b_i < b->size[0]; b_i++) {
            zik->data[b_i] = b->data[b_i];
          }
          profileLoopEnd();
          solve_from_lu_di(b_N, b_S, (double *)&zik->data[0], b->size[0]);
          cs_di_sfree(b_S);
          cs_di_nfree(b_N);
        }
      } else {
        cs_di *cxA;
        cs_din *N;
        cs_dis *S;
        if (thism < cidxInt->data[cidxInt->size[0] - 1]) {
          nvtxMarkA("#sparse_ctranspose#" MW_AT_LINE);
          sparse_ctranspose(y_d, y_colidx, y_rowidx, thism,
                            cidxInt->data[cidxInt->size[0] - 1], in_d,
                            in_colidx, in_rowidx, &n, &i);
          cxA = makeCXSparseMatrix(in_colidx->data[in_colidx->size[0] - 1] - 1,
                                   i, n, &in_colidx->data[0],
                                   &in_rowidx->data[0], &in_d->data[0]);
        } else {
          cxA = makeCXSparseMatrix(y_colidx->data[y_colidx->size[0] - 1] - 1,
                                   cidxInt->data[cidxInt->size[0] - 1], thism,
                                   &y_colidx->data[0], &y_rowidx->data[0],
                                   &y_d->data[0]);
        }
        S = cs_di_sqr(2, cxA, 1);
        N = cs_di_qr(cxA, S);
        cs_di_spfree(cxA);
        n = qr_rank_di(N, &tol);
        if (thism > cidxInt->data[cidxInt->size[0] - 1]) {
          thism = cidxInt->data[cidxInt->size[0] - 1];
        }
        if (n < thism) {
          char_T str[14];
          y = nullptr;
          m = emlrtCreateCharArray(2, &iv[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 6, m, &rfmt[0]);
          emlrtAssign(&y, m);
          b_y = nullptr;
          m1 = emlrtCreateDoubleScalar(tol);
          emlrtAssign(&b_y, m1);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(y, b_y, &d_emlrtMCI),
                           "<output of sprintf>", str);
          nvtxMarkA("#warning#" MW_AT_LINE);
          internal::warning(n, str);
        }
        b_i = zik->size[0];
        zik->size[0] = cidxInt->data[cidxInt->size[0] - 1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(zik, b_i, &bd_emlrtRTEI);
        if (b->size[0] < cidxInt->data[cidxInt->size[0] - 1]) {
          b_i = outBuff->size[0];
          outBuff->size[0] = cidxInt->data[cidxInt->size[0] - 1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(outBuff, b_i, &dd_emlrtRTEI);
        } else {
          b_i = outBuff->size[0];
          outBuff->size[0] = b->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(outBuff, b_i, &cd_emlrtRTEI);
        }
        k = b->size[0];
        profileLoopStart("getCoeffsAndInitialConditions_loop_38", __LINE__,
                         (k - 1) + 1, "");
        for (b_i = 0; b_i < k; b_i++) {
          outBuff->data[b_i] = b->data[b_i];
        }
        profileLoopEnd();
        solve_from_qr_di(N, S, (double *)&outBuff->data[0], b->size[0],
                         cidxInt->data[cidxInt->size[0] - 1]);
        n = cidxInt->data[cidxInt->size[0] - 1];
        profileLoopStart("getCoeffsAndInitialConditions_loop_39", __LINE__,
                         (n - 1) + 1, "");
        for (b_i = 0; b_i < n; b_i++) {
          zik->data[b_i] = outBuff->data[b_i];
        }
        profileLoopEnd();
        cs_di_sfree(S);
        cs_di_nfree(N);
      }
      n = zi->size[0];
      profileLoopStart("getCoeffsAndInitialConditions_loop_36", __LINE__,
                       (n - 1) + 1, "");
      for (b_i = 0; b_i < n; b_i++) {
        zi->data[b_i + zi->size[0] * ns] =
            static_cast<real32_T>(zik->data[b_i]);
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_outBuff);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&outBuff);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&c_in_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&c_in_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_in_d);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_in_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_in_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_in_d);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&in_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&in_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&in_d);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&zik);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_t);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&t);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&sortedIndices);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&cidxInt);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&ridxInt);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&y_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&y_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&y_d);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&yf);
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&colIdx);
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&rowIdx);
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&rhs);
  } else {
    zi->size[0] = 0;
    zi->size[1] = static_cast<int32_T>(numStage);
  }
  *Letr = std::fmax(1.0F, 3.0F * ord);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

} // namespace coder
static int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  nvtxRangePushA("#fcn#div_s32#" MW_AT_LOCATION);
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  nvtxRangePop();
  return quotient;
}

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[14])
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[14])
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

//
//
namespace coder {
void filtfilt(const emxArray_real32_T *ctf, const emxArray_real32_T *varargin_1,
              const emxArray_real32_T *varargin_2, emxArray_real32_T *y)
{
  emxArray_real32_T b_gpu_A;
  emxArray_real32_T b_gpu_B;
  emxArray_real32_T b_gpu_xt;
  emxArray_real32_T b_gpu_yc2;
  emxArray_real32_T b_gpu_yc3;
  emxArray_real32_T b_gpu_yc5;
  emxArray_real32_T b_gpu_ytemp;
  emxArray_real32_T b_gpu_zi;
  emxArray_real32_T b_gpu_zo;
  emxArray_real32_T c_gpu_A;
  emxArray_real32_T c_gpu_B;
  emxArray_real32_T c_gpu_xt;
  emxArray_real32_T c_gpu_yc2;
  emxArray_real32_T c_gpu_yc3;
  emxArray_real32_T c_gpu_ytemp;
  emxArray_real32_T c_gpu_zi;
  emxArray_real32_T c_gpu_zo;
  emxArray_real32_T d_gpu_A;
  emxArray_real32_T d_gpu_B;
  emxArray_real32_T d_gpu_yc2;
  emxArray_real32_T d_gpu_yc3;
  emxArray_real32_T d_gpu_zi;
  emxArray_real32_T d_gpu_zo;
  emxArray_real32_T e_gpu_A;
  emxArray_real32_T e_gpu_B;
  emxArray_real32_T e_gpu_zi;
  emxArray_real32_T e_gpu_zo;
  emxArray_real32_T f_gpu_A;
  emxArray_real32_T f_gpu_B;
  emxArray_real32_T f_gpu_zi;
  emxArray_real32_T f_gpu_zo;
  emxArray_real32_T g_gpu_A;
  emxArray_real32_T g_gpu_B;
  emxArray_real32_T gpu_A;
  emxArray_real32_T gpu_B;
  emxArray_real32_T gpu__1;
  emxArray_real32_T gpu__2;
  emxArray_real32_T gpu__3;
  emxArray_real32_T gpu__4;
  emxArray_real32_T gpu_r2;
  emxArray_real32_T gpu_xc1;
  emxArray_real32_T gpu_xt;
  emxArray_real32_T gpu_y;
  emxArray_real32_T gpu_yc2;
  emxArray_real32_T gpu_yc3;
  emxArray_real32_T gpu_yc5;
  emxArray_real32_T gpu_ytemp;
  emxArray_real32_T gpu_zi;
  emxArray_real32_T gpu_zo;
  emxArray_real32_T h_gpu_A;
  emxArray_real32_T h_gpu_B;
  emxArray_real32_T i_gpu_A;
  emxArray_real32_T i_gpu_B;
  emxArray_real32_T j_gpu_A;
  emxArray_real32_T j_gpu_B;
  emxArray_real32_T k_gpu_A;
  emxArray_real32_T k_gpu_B;
  emxArray_real32_T l_gpu_A;
  emxArray_real32_T l_gpu_B;
  emxArray_real32_T *A;
  emxArray_real32_T *B;
  emxArray_real32_T *X;
  emxArray_real32_T *b_cpu_A;
  emxArray_real32_T *b_cpu_B;
  emxArray_real32_T *b_cpu_xt;
  emxArray_real32_T *b_cpu_yc2;
  emxArray_real32_T *b_cpu_yc3;
  emxArray_real32_T *b_cpu_yc5;
  emxArray_real32_T *b_cpu_ytemp;
  emxArray_real32_T *b_cpu_zi;
  emxArray_real32_T *b_cpu_zo;
  emxArray_real32_T *b_xc1;
  emxArray_real32_T *c_cpu_A;
  emxArray_real32_T *c_cpu_B;
  emxArray_real32_T *c_cpu_xt;
  emxArray_real32_T *c_cpu_yc2;
  emxArray_real32_T *c_cpu_yc3;
  emxArray_real32_T *c_cpu_ytemp;
  emxArray_real32_T *c_cpu_zi;
  emxArray_real32_T *c_cpu_zo;
  emxArray_real32_T *cpu_A;
  emxArray_real32_T *cpu_B;
  emxArray_real32_T *cpu__1;
  emxArray_real32_T *cpu__2;
  emxArray_real32_T *cpu__3;
  emxArray_real32_T *cpu__4;
  emxArray_real32_T *cpu_r2;
  emxArray_real32_T *cpu_xc1;
  emxArray_real32_T *cpu_xt;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *cpu_yc2;
  emxArray_real32_T *cpu_yc3;
  emxArray_real32_T *cpu_yc5;
  emxArray_real32_T *cpu_ytemp;
  emxArray_real32_T *cpu_zi;
  emxArray_real32_T *cpu_zo;
  emxArray_real32_T *d_cpu_A;
  emxArray_real32_T *d_cpu_B;
  emxArray_real32_T *d_cpu_yc2;
  emxArray_real32_T *d_cpu_yc3;
  emxArray_real32_T *d_cpu_zi;
  emxArray_real32_T *d_cpu_zo;
  emxArray_real32_T *den;
  emxArray_real32_T *e_cpu_A;
  emxArray_real32_T *e_cpu_B;
  emxArray_real32_T *e_cpu_zi;
  emxArray_real32_T *e_cpu_zo;
  emxArray_real32_T *f_cpu_A;
  emxArray_real32_T *f_cpu_B;
  emxArray_real32_T *f_cpu_zi;
  emxArray_real32_T *f_cpu_zo;
  emxArray_real32_T *g_cpu_A;
  emxArray_real32_T *g_cpu_B;
  emxArray_real32_T *h_cpu_A;
  emxArray_real32_T *h_cpu_B;
  emxArray_real32_T *i_cpu_A;
  emxArray_real32_T *i_cpu_B;
  emxArray_real32_T *j_cpu_A;
  emxArray_real32_T *j_cpu_B;
  emxArray_real32_T *k_cpu_A;
  emxArray_real32_T *k_cpu_B;
  emxArray_real32_T *l_cpu_A;
  emxArray_real32_T *l_cpu_B;
  emxArray_real32_T *num;
  emxArray_real32_T *r;
  emxArray_real32_T *r1;
  emxArray_real32_T *xc1;
  emxArray_real32_T *xt;
  emxArray_real32_T *ytemp;
  emxArray_real32_T *zi;
  real_T P;
  real_T Q;
  real_T numStage;
  int32_T i;
  real32_T Letr;
  boolean_T A_outdatedOnCpu;
  boolean_T A_outdatedOnGpu;
  boolean_T B_outdatedOnCpu;
  boolean_T B_outdatedOnGpu;
  boolean_T a__2_outdatedOnGpu;
  boolean_T a__4_outdatedOnGpu;
  boolean_T b_zo_outdatedOnCpu;
  boolean_T b_zo_outdatedOnGpu;
  boolean_T r2_outdatedOnCpu;
  boolean_T xIsRow;
  boolean_T xt_outdatedOnCpu;
  boolean_T xt_outdatedOnGpu;
  boolean_T yc2_outdatedOnCpu;
  boolean_T yc3_outdatedOnGpu;
  boolean_T yc5_outdatedOnCpu;
  boolean_T yc5_outdatedOnGpu;
  boolean_T zi_outdatedOnCpu;
  boolean_T zi_outdatedOnGpu;
  boolean_T zo_outdatedOnCpu;
  boolean_T zo_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#filtfilt#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_ytemp);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_r2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&g_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&g_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&l_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&l_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&f_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&f_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&f_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&k_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&k_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_xt);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&e_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&e_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&j_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&j_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_zi);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_xt);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&i_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&i_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&e_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&h_gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&h_gpu_B);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_yc5);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_yc5);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu__2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&f_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu__4);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_ytemp);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&e_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu__1);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu__3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_xt);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_xc1);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_ytemp);
  r2_outdatedOnCpu = false;
  yc3_outdatedOnGpu = false;
  b_zo_outdatedOnCpu = false;
  b_zo_outdatedOnGpu = false;
  yc2_outdatedOnCpu = false;
  a__2_outdatedOnGpu = false;
  zo_outdatedOnCpu = false;
  zo_outdatedOnGpu = false;
  yc5_outdatedOnGpu = false;
  yc5_outdatedOnCpu = false;
  xt_outdatedOnCpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&X, 3, &b_emlrtRTEI, true);
  i = X->size[0] * X->size[1] * X->size[2];
  X->size[0] = varargin_2->size[0];
  X->size[1] = varargin_2->size[1];
  X->size[2] = varargin_2->size[2];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(X, i, &emlrtRTEI);
  profileLoopStart(
      "filtfilt_loop_0", __LINE__,
      (varargin_2->size[0] * varargin_2->size[1] * varargin_2->size[2] - 1) + 1,
      "1553");
  for (i = 0;
       i < varargin_2->size[0] * varargin_2->size[1] * varargin_2->size[2];
       i++) {
    X->data[i] = varargin_2->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&num, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&den, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#filtfiltParser#" MW_AT_LINE);
  filtfiltParser(ctf, varargin_1, num, den, &numStage, &P, &Q, &xIsRow,
                 &yc5_outdatedOnGpu);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&B, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&A, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&zi, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_ytemp, 1, &wb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_xc1, 2, &xb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_xt, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&xt, 1, &j_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu__3, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_zo, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu__1, 1, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_zo, 1, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_yc2, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_zo, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_yc2, 1, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_zo, 1, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_ytemp, 1, &ac_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_yc3, 2, &bc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_yc3, 1, &cc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu__4, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&e_cpu_zo, 2, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu__2, 1, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f_cpu_zo, 1, &vb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_yc5, 2, &dc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_yc5, 1, &ec_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&r, 2, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&xc1, 2, &i_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_B, 2, &l_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_A, 2, &n_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_zi, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_B, 2, &t_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_A, 2, &u_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_B, 2, &m_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_A, 2, &o_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_xt, 1, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_zi, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_B, 2, &v_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_A, 2, &x_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_y, 1, &ab_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&e_cpu_B, 2, &p_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&e_cpu_A, 2, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_zi, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&ytemp, 1, &w_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&r1, 2, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_xc1, 2, &cb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f_cpu_B, 2, &bb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f_cpu_A, 2, &db_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_zi, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&g_cpu_B, 2, &eb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&g_cpu_A, 2, &fb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&h_cpu_B, 2, &jb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&h_cpu_A, 2, &lb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_xt, 1, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&i_cpu_B, 2, &ib_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&i_cpu_A, 2, &kb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_yc3, 2, &mb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&e_cpu_zi, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&j_cpu_B, 2, &nb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&j_cpu_A, 2, &pb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_yc3, 1, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f_cpu_zi, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&k_cpu_B, 2, &ob_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&k_cpu_A, 2, &qb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_yc2, 2, &rb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&l_cpu_B, 2, &sb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&l_cpu_A, 2, &ub_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_yc2, 1, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_r2, 2, &fc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_ytemp, 1, &gb_emlrtRTEI, true);
  if ((!xIsRow) || (!yc5_outdatedOnGpu) ||
      ((X->size[0] == 0) || (X->size[1] == 0) || (X->size[2] == 0))) {
    y->size[0] = 0;
    y->size[1] = 0;
    y->size[2] = 1;
  } else {
    if ((X->size[2] == 1) && (X->size[0] == 1)) {
      i = X->size[0] * X->size[1] * X->size[2];
      X->size[0] = X->size[1];
      X->size[1] = 1;
      X->size[2] = 1;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(X, i, &b_emlrtRTEI);
    } else {
      xIsRow = false;
    }
    nvtxMarkA("#getCoeffsAndInitialConditions#" MW_AT_LINE);
    getCoeffsAndInitialConditions(num, den, numStage, P, Q, B, A, zi, &Letr);
    if (X->size[1] * X->size[2] == 1) {
      if (X->size[0] < 10000) {
        int32_T b_i;
        int32_T i1;
        real32_T Y_data[9999];
        yc5_outdatedOnGpu = false;
        a__2_outdatedOnGpu = false;
        b_i = X->size[0];
        profileLoopStart("filtfilt_loop_1", __LINE__, (X->size[0] - 1) + 1,
                         "1554");
        for (i = 0; i < X->size[0]; i++) {
          Y_data[i] = X->data[i];
        }
        profileLoopEnd();
        i1 = static_cast<int32_T>(numStage);
        profileLoopStart("filtfilt_loop_4", __LINE__, (i1 - 1) + 1, "1555");
        for (int32_T ii{0}; ii < i1; ii++) {
          int32_T b_loop_ub;
          int32_T i7;
          int32_T i9;
          int32_T loop_ub;
          real32_T b_y;
          real32_T f1;
          f1 = static_cast<real32_T>(b_i) - Letr;
          if (f1 > static_cast<real32_T>(b_i) - 1.0F) {
            loop_ub = 0;
            i7 = 1;
            i9 = -1;
          } else {
            loop_ub = b_i - 2;
            i7 = -1;
            i9 = static_cast<int32_T>(f1) - 1;
          }
          f1 = 2.0F * Y_data[0];
          b_y = 2.0F * Y_data[b_i - 1];
          yc5_outdatedOnCpu = false;
          i = b_cpu_ytemp->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          b_cpu_ytemp->size[0] = (static_cast<int32_T>(Letr + 1.0F) + b_i) +
                                 div_s32(i9 - loop_ub, i7);
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_cpu_ytemp, i, &k_emlrtRTEI);
          b_loop_ub = static_cast<int32_T>(Letr + 1.0F) - 2;
          profileLoopStart("filtfilt_loop_11", __LINE__, b_loop_ub + 1, "1556");
          for (i = 0; i <= b_loop_ub; i++) {
            b_cpu_ytemp->data[i] =
                f1 - Y_data[(static_cast<int32_T>(Letr + 1.0F) - i) - 1];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          profileLoopStart("filtfilt_loop_14", __LINE__, (b_i - 1) + 1, "1557");
          for (i = 0; i < b_i; i++) {
            b_cpu_ytemp->data[(i + static_cast<int32_T>(Letr + 1.0F)) - 1] =
                Y_data[i];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          b_loop_ub = div_s32(i9 - loop_ub, i7);
          profileLoopStart("filtfilt_loop_15", __LINE__, b_loop_ub + 1, "1558");
          for (i = 0; i <= b_loop_ub; i++) {
            b_cpu_ytemp
                ->data[((i + static_cast<int32_T>(Letr + 1.0F)) + b_i) - 1] =
                b_y - Y_data[loop_ub + i7 * i];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = e_cpu_B->size[0] * e_cpu_B->size[1];
          e_cpu_B->size[0] = 1;
          e_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(e_cpu_B, i, &p_emlrtRTEI);
          profileLoopStart("filtfilt_loop_18", __LINE__, (B->size[1] - 1) + 1,
                           "1559");
          for (i = 0; i < B->size[1]; i++) {
            e_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = e_cpu_A->size[0] * e_cpu_A->size[1];
          e_cpu_A->size[0] = 1;
          e_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(e_cpu_A, i, &s_emlrtRTEI);
          profileLoopStart("filtfilt_loop_21", __LINE__, (A->size[1] - 1) + 1,
                           "1560");
          for (i = 0; i < A->size[1]; i++) {
            e_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = c_cpu_zi->size[0];
          c_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(c_cpu_zi, i, &e_emlrtRTEI);
          profileLoopStart("filtfilt_loop_24", __LINE__, (zi->size[0] - 1) + 1,
                           "1561");
          for (i = 0; i < zi->size[0]; i++) {
            c_cpu_zi->data[i] =
                zi->data[i + zi->size[0] * ii] * b_cpu_ytemp->data[0];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(e_cpu_B, &B_outdatedOnCpu, &gpu_B, &B_outdatedOnGpu, e_cpu_A,
                 &A_outdatedOnCpu, &gpu_A, &A_outdatedOnGpu, b_cpu_ytemp,
                 &gpu_ytemp, &yc5_outdatedOnCpu, c_cpu_zi, &zi_outdatedOnCpu,
                 &gpu_zi, &zi_outdatedOnGpu, cpu_ytemp, &yc5_outdatedOnGpu,
                 &b_gpu_ytemp, &a__2_outdatedOnGpu);
          loop_ub = cpu_ytemp->size[0] - 1;
          i = ytemp->size[0];
          ytemp->size[0] = cpu_ytemp->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(ytemp, i, &w_emlrtRTEI);
          profileLoopStart("filtfilt_loop_27", __LINE__, loop_ub + 1, "1562");
          for (i = 0; i <= loop_ub; i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_ytemp, &b_gpu_ytemp);
            }
            yc5_outdatedOnGpu = false;
            ytemp->data[i] = cpu_ytemp->data[loop_ub - i];
          }
          profileLoopEnd();
          yc5_outdatedOnGpu = false;
          a__2_outdatedOnGpu = false;
          i = cpu_ytemp->size[0];
          cpu_ytemp->size[0] = ytemp->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(cpu_ytemp, i, &y_emlrtRTEI);
          profileLoopStart("filtfilt_loop_30", __LINE__,
                           (ytemp->size[0] - 1) + 1, "1563");
          for (i = 0; i < ytemp->size[0]; i++) {
            cpu_ytemp->data[i] = ytemp->data[i];
            a__2_outdatedOnGpu = true;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = f_cpu_B->size[0] * f_cpu_B->size[1];
          f_cpu_B->size[0] = 1;
          f_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(f_cpu_B, i, &bb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_33", __LINE__, (B->size[1] - 1) + 1,
                           "1564");
          for (i = 0; i < B->size[1]; i++) {
            f_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = f_cpu_A->size[0] * f_cpu_A->size[1];
          f_cpu_A->size[0] = 1;
          f_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(f_cpu_A, i, &db_emlrtRTEI);
          profileLoopStart("filtfilt_loop_35", __LINE__, (A->size[1] - 1) + 1,
                           "1565");
          for (i = 0; i < A->size[1]; i++) {
            f_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = d_cpu_zi->size[0];
          d_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(d_cpu_zi, i, &e_emlrtRTEI);
          profileLoopStart("filtfilt_loop_37", __LINE__, (zi->size[0] - 1) + 1,
                           "1566");
          for (i = 0; i < zi->size[0]; i++) {
            d_cpu_zi->data[i] =
                zi->data[i + zi->size[0] * ii] * cpu_ytemp->data[0];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          yc5_outdatedOnCpu = false;
          i = c_cpu_ytemp->size[0];
          c_cpu_ytemp->size[0] = cpu_ytemp->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(c_cpu_ytemp, i, &gb_emlrtRTEI);
          b_loop_ub = cpu_ytemp->size[0] - 1;
          profileLoopStart("filtfilt_loop_41", __LINE__, b_loop_ub + 1, "1567");
          for (i = 0; i <= b_loop_ub; i++) {
            c_cpu_ytemp->data[i] = cpu_ytemp->data[i];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(f_cpu_B, &B_outdatedOnCpu, &b_gpu_B, &B_outdatedOnGpu, f_cpu_A,
                 &A_outdatedOnCpu, &b_gpu_A, &A_outdatedOnGpu, c_cpu_ytemp,
                 &c_gpu_ytemp, &yc5_outdatedOnCpu, d_cpu_zi, &zi_outdatedOnCpu,
                 &b_gpu_zi, &zi_outdatedOnGpu, cpu_ytemp, &yc5_outdatedOnGpu,
                 &b_gpu_ytemp, &a__2_outdatedOnGpu);
          f1 = static_cast<real32_T>(cpu_ytemp->size[0]) - Letr;
          if (Letr + 1.0F > f1) {
            loop_ub = 1;
            i7 = 1;
          } else {
            loop_ub = static_cast<int32_T>(f1);
            i7 = -1;
          }
          profileLoopStart("filtfilt_loop_45", __LINE__, (b_i - 1) + 1, "1568");
          for (i = 0; i < b_i; i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_ytemp, &b_gpu_ytemp);
            }
            yc5_outdatedOnGpu = false;
            Y_data[i] = cpu_ytemp->data[(loop_ub + i7 * i) - 1];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        i = y->size[0] * y->size[1] * y->size[2];
        y->size[0] = b_i;
        y->size[1] = 1;
        y->size[2] = 1;
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(y, i, &h_emlrtRTEI);
        profileLoopStart("filtfilt_loop_8", __LINE__, (b_i - 1) + 1, "1569");
        for (i = 0; i < b_i; i++) {
          y->data[i] = Y_data[i];
        }
        profileLoopEnd();
      } else {
        int32_T b_i;
        i = y->size[0] * y->size[1] * y->size[2];
        y->size[0] = X->size[0];
        y->size[1] = X->size[1];
        y->size[2] = X->size[2];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(y, i, &c_emlrtRTEI);
        profileLoopStart("filtfilt_loop_2", __LINE__,
                         (X->size[0] * X->size[1] * X->size[2] - 1) + 1,
                         "1570");
        for (i = 0; i < X->size[0] * X->size[1] * X->size[2]; i++) {
          y->data[i] = X->data[i];
        }
        profileLoopEnd();
        b_i = static_cast<int32_T>(numStage);
        profileLoopStart("filtfilt_loop_5", __LINE__, (b_i - 1) + 1, "1571");
        for (int32_T ii{0}; ii < b_i; ii++) {
          int32_T b_i1;
          int32_T b_loop_ub;
          int32_T i7;
          int32_T i9;
          int32_T loop_ub;
          real32_T f1;
          f1 = 2.0F * y->data[0];
          i = static_cast<int32_T>(Letr + 1.0F) - 1;
          b_i1 = xt->size[0];
          xt->size[0] = static_cast<int32_T>(Letr + 1.0F) - 1;
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(xt, b_i1, &j_emlrtRTEI);
          profileLoopStart("filtfilt_loop_7", __LINE__, (i - 1) + 1, "1572");
          for (b_i1 = 0; b_i1 < i; b_i1++) {
            xt->data[b_i1] =
                -y->data[(static_cast<int32_T>(Letr + 1.0F) - b_i1) - 1] + f1;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = c_cpu_B->size[0] * c_cpu_B->size[1];
          c_cpu_B->size[0] = 1;
          c_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(c_cpu_B, i, &m_emlrtRTEI);
          profileLoopStart("filtfilt_loop_12", __LINE__, (B->size[1] - 1) + 1,
                           "1573");
          for (i = 0; i < B->size[1]; i++) {
            c_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = c_cpu_A->size[0] * c_cpu_A->size[1];
          c_cpu_A->size[0] = 1;
          c_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(c_cpu_A, i, &o_emlrtRTEI);
          profileLoopStart("filtfilt_loop_16", __LINE__, (A->size[1] - 1) + 1,
                           "1574");
          for (i = 0; i < A->size[1]; i++) {
            c_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          xt_outdatedOnGpu = false;
          i = b_cpu_xt->size[0];
          b_cpu_xt->size[0] = xt->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_cpu_xt, i, &r_emlrtRTEI);
          profileLoopStart("filtfilt_loop_19", __LINE__, (xt->size[0] - 1) + 1,
                           "1575");
          for (i = 0; i < xt->size[0]; i++) {
            b_cpu_xt->data[i] = xt->data[i];
            xt_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = b_cpu_zi->size[0];
          b_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_cpu_zi, i, &e_emlrtRTEI);
          profileLoopStart("filtfilt_loop_22", __LINE__, (zi->size[0] - 1) + 1,
                           "1576");
          for (i = 0; i < zi->size[0]; i++) {
            b_cpu_zi->data[i] = zi->data[i + zi->size[0] * ii] * xt->data[0];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#b_filter#" MW_AT_LINE);
          b_filter(c_cpu_B, &B_outdatedOnCpu, &c_gpu_B, &B_outdatedOnGpu,
                   c_cpu_A, &A_outdatedOnCpu, &c_gpu_A, &A_outdatedOnGpu,
                   b_cpu_xt, &gpu_xt, &xt_outdatedOnGpu, b_cpu_zi,
                   &zi_outdatedOnCpu, &c_gpu_zi, &zi_outdatedOnGpu, cpu__1,
                   &yc5_outdatedOnGpu, &gpu__1, &a__2_outdatedOnGpu, b_cpu_zo,
                   &zo_outdatedOnCpu, &gpu_zo, &zo_outdatedOnGpu);
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = d_cpu_B->size[0] * d_cpu_B->size[1];
          d_cpu_B->size[0] = 1;
          d_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(d_cpu_B, i, &v_emlrtRTEI);
          profileLoopStart("filtfilt_loop_26", __LINE__, (B->size[1] - 1) + 1,
                           "1577");
          for (i = 0; i < B->size[1]; i++) {
            d_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = d_cpu_A->size[0] * d_cpu_A->size[1];
          d_cpu_A->size[0] = 1;
          d_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(d_cpu_A, i, &x_emlrtRTEI);
          profileLoopStart("filtfilt_loop_29", __LINE__, (A->size[1] - 1) + 1,
                           "1578");
          for (i = 0; i < A->size[1]; i++) {
            d_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          i = y->size[0];
          yc5_outdatedOnCpu = false;
          b_i1 = cpu_y->size[0];
          cpu_y->size[0] = y->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(cpu_y, b_i1, &ab_emlrtRTEI);
          profileLoopStart("filtfilt_loop_32", __LINE__, (i - 1) + 1, "1579");
          for (b_i1 = 0; b_i1 < i; b_i1++) {
            cpu_y->data[b_i1] = y->data[b_i1];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#b_filter#" MW_AT_LINE);
          b_filter(d_cpu_B, &B_outdatedOnCpu, &d_gpu_B, &B_outdatedOnGpu,
                   d_cpu_A, &A_outdatedOnCpu, &d_gpu_A, &A_outdatedOnGpu, cpu_y,
                   &gpu_y, &yc5_outdatedOnCpu, b_cpu_zo, &zo_outdatedOnCpu,
                   &gpu_zo, &zo_outdatedOnGpu, b_cpu_yc2, &yc2_outdatedOnCpu,
                   &gpu_yc2, &a__2_outdatedOnGpu, d_cpu_zo, &b_zo_outdatedOnCpu,
                   &b_gpu_zo, &b_zo_outdatedOnGpu);
          f1 = static_cast<real32_T>(y->size[0]) - Letr;
          if (f1 >
              static_cast<real32_T>(static_cast<real_T>(y->size[0]) - 1.0)) {
            loop_ub = 0;
            i7 = 1;
            i9 = -1;
          } else {
            loop_ub = static_cast<int32_T>(static_cast<real32_T>(
                          static_cast<real_T>(y->size[0]) - 1.0)) -
                      1;
            i7 = -1;
            i9 = static_cast<int32_T>(f1) - 1;
          }
          f1 = 2.0F * y->data[y->size[0] - 1];
          i = xt->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          xt->size[0] = div_s32(i9 - loop_ub, i7) + 1;
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(xt, i, &hb_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_34", __LINE__,
                           div_s32(i9 - loop_ub, i7) + 1, "1580");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (i = 0; i <= div_s32(i9 - loop_ub, i7); i++) {
            xt->data[i] = -y->data[loop_ub + i7 * i] + f1;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = h_cpu_B->size[0] * h_cpu_B->size[1];
          h_cpu_B->size[0] = 1;
          h_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(h_cpu_B, i, &jb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_39", __LINE__, (B->size[1] - 1) + 1,
                           "1581");
          for (i = 0; i < B->size[1]; i++) {
            h_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = h_cpu_A->size[0] * h_cpu_A->size[1];
          h_cpu_A->size[0] = 1;
          h_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(h_cpu_A, i, &lb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_42", __LINE__, (A->size[1] - 1) + 1,
                           "1582");
          for (i = 0; i < A->size[1]; i++) {
            h_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          xt_outdatedOnGpu = false;
          i = c_cpu_xt->size[0];
          c_cpu_xt->size[0] = xt->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(c_cpu_xt, i, &r_emlrtRTEI);
          profileLoopStart("filtfilt_loop_44", __LINE__, (xt->size[0] - 1) + 1,
                           "1583");
          for (i = 0; i < xt->size[0]; i++) {
            c_cpu_xt->data[i] = xt->data[i];
            xt_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(h_cpu_B, &B_outdatedOnCpu, &e_gpu_B, &B_outdatedOnGpu, h_cpu_A,
                 &A_outdatedOnCpu, &e_gpu_A, &A_outdatedOnGpu, c_cpu_xt,
                 &b_gpu_xt, &xt_outdatedOnGpu, d_cpu_zo, &b_zo_outdatedOnCpu,
                 &b_gpu_zo, &b_zo_outdatedOnGpu, b_cpu_yc3, &yc5_outdatedOnGpu,
                 &gpu_yc3, &yc3_outdatedOnGpu);
          if (b_cpu_yc3->size[0] < 1) {
            loop_ub = 0;
            i7 = 1;
            i9 = -1;
          } else {
            loop_ub = b_cpu_yc3->size[0] - 1;
            i7 = -1;
            i9 = 0;
          }
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = j_cpu_B->size[0] * j_cpu_B->size[1];
          j_cpu_B->size[0] = 1;
          j_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(j_cpu_B, i, &nb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_46", __LINE__, (B->size[1] - 1) + 1,
                           "1584");
          for (i = 0; i < B->size[1]; i++) {
            j_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = j_cpu_A->size[0] * j_cpu_A->size[1];
          j_cpu_A->size[0] = 1;
          j_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(j_cpu_A, i, &pb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_48", __LINE__, (A->size[1] - 1) + 1,
                           "1585");
          for (i = 0; i < A->size[1]; i++) {
            j_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          yc5_outdatedOnCpu = false;
          yc3_outdatedOnGpu = false;
          i = d_cpu_yc3->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          d_cpu_yc3->size[0] = div_s32(i9 - loop_ub, i7) + 1;
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(d_cpu_yc3, i, &r_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_50", __LINE__,
                           div_s32(i9 - loop_ub, i7) + 1, "1586");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (i = 0; i <= div_s32(i9 - loop_ub, i7); i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_yc3, &gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            d_cpu_yc3->data[i] = b_cpu_yc3->data[loop_ub + i7 * i];
            yc3_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = f_cpu_zi->size[0];
          f_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(f_cpu_zi, i, &e_emlrtRTEI);
          profileLoopStart("filtfilt_loop_52", __LINE__, (zi->size[0] - 1) + 1,
                           "1587");
          for (i = 0; i < zi->size[0]; i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_yc3, &gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            f_cpu_zi->data[i] = zi->data[i + zi->size[0] * ii] *
                                b_cpu_yc3->data[b_cpu_yc3->size[0] - 1];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#b_filter#" MW_AT_LINE);
          b_filter(j_cpu_B, &B_outdatedOnCpu, &f_gpu_B, &B_outdatedOnGpu,
                   j_cpu_A, &A_outdatedOnCpu, &f_gpu_A, &A_outdatedOnGpu,
                   d_cpu_yc3, &b_gpu_yc3, &yc3_outdatedOnGpu, f_cpu_zi,
                   &zi_outdatedOnCpu, &d_gpu_zi, &zi_outdatedOnGpu, cpu__2,
                   &yc5_outdatedOnGpu, &gpu__2, &a__2_outdatedOnGpu, f_cpu_zo,
                   &b_zo_outdatedOnCpu, &c_gpu_zo, &b_zo_outdatedOnGpu);
          if (b_cpu_yc2->size[0] < 1) {
            loop_ub = 0;
            i7 = 1;
            i9 = -1;
          } else {
            loop_ub = b_cpu_yc2->size[0] - 1;
            i7 = -1;
            i9 = 0;
          }
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = l_cpu_B->size[0] * l_cpu_B->size[1];
          l_cpu_B->size[0] = 1;
          l_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(l_cpu_B, i, &sb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_56", __LINE__, (B->size[1] - 1) + 1,
                           "1588");
          for (i = 0; i < B->size[1]; i++) {
            l_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = l_cpu_A->size[0] * l_cpu_A->size[1];
          l_cpu_A->size[0] = 1;
          l_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(l_cpu_A, i, &ub_emlrtRTEI);
          profileLoopStart("filtfilt_loop_58", __LINE__, (A->size[1] - 1) + 1,
                           "1589");
          for (i = 0; i < A->size[1]; i++) {
            l_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          a__2_outdatedOnGpu = false;
          i = d_cpu_yc2->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          d_cpu_yc2->size[0] = div_s32(i9 - loop_ub, i7) + 1;
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(d_cpu_yc2, i, &r_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_59", __LINE__,
                           div_s32(i9 - loop_ub, i7) + 1, "1590");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (i = 0; i <= div_s32(i9 - loop_ub, i7); i++) {
            if (yc2_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_yc2, &gpu_yc2);
            }
            yc2_outdatedOnCpu = false;
            d_cpu_yc2->data[i] = b_cpu_yc2->data[loop_ub + i7 * i];
            a__2_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(l_cpu_B, &B_outdatedOnCpu, &g_gpu_B, &B_outdatedOnGpu, l_cpu_A,
                 &A_outdatedOnCpu, &g_gpu_A, &A_outdatedOnGpu, d_cpu_yc2,
                 &b_gpu_yc2, &a__2_outdatedOnGpu, f_cpu_zo, &b_zo_outdatedOnCpu,
                 &c_gpu_zo, &b_zo_outdatedOnGpu, b_cpu_yc5, &yc5_outdatedOnCpu,
                 &gpu_yc5, &yc5_outdatedOnGpu);
          if (b_cpu_yc5->size[0] < 1) {
            i7 = 0;
            i9 = 1;
            loop_ub = -1;
          } else {
            i7 = b_cpu_yc5->size[0] - 1;
            i9 = -1;
            loop_ub = 0;
          }
          i = y->size[0] * y->size[1] * y->size[2];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          y->size[0] = div_s32(loop_ub - i7, i9) + 1;
          y->size[1] = 1;
          y->size[2] = 1;
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(y, i, &c_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          b_loop_ub = div_s32(loop_ub - i7, i9) + 1;
          profileLoopStart("filtfilt_loop_62", __LINE__, (b_loop_ub - 1) + 1,
                           "1591");
          for (i = 0; i < b_loop_ub; i++) {
            if (yc5_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_yc5, &gpu_yc5);
            }
            yc5_outdatedOnCpu = false;
            y->data[i] = b_cpu_yc5->data[i7 + i9 * i];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
      }
    } else {
      int32_T i1;
      uint32_T sz_idx_0;
      uint32_T sz_idx_1;
      uint32_T sz_idx_2;
      sz_idx_0 = static_cast<uint32_T>(X->size[0]);
      sz_idx_1 = static_cast<uint32_T>(X->size[1]);
      sz_idx_2 = static_cast<uint32_T>(X->size[2]);
      a__4_outdatedOnGpu = false;
      yc3_outdatedOnGpu = false;
      i = cpu_xc1->size[0] * cpu_xc1->size[1];
      cpu_xc1->size[0] = X->size[0];
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      cpu_xc1->size[1] =
          div_s32(X->size[0] * X->size[1] * X->size[2], X->size[0]);
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_xc1, i, &d_emlrtRTEI);
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      profileLoopStart("filtfilt_loop_3", __LINE__,
                       (static_cast<int32_T>(sz_idx_0) *
                            div_s32(X->size[0] * X->size[1] * X->size[2],
                                    static_cast<int32_T>(sz_idx_0)) -
                        1) +
                           1,
                       "1592");
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      for (i = 0; i < static_cast<int32_T>(sz_idx_0) *
                          div_s32(X->size[0] * X->size[1] * X->size[2],
                                  static_cast<int32_T>(sz_idx_0));
           i++) {
        cpu_xc1->data[i] = X->data[i];
        yc3_outdatedOnGpu = true;
      }
      profileLoopEnd();
      i1 = static_cast<int32_T>(numStage);
      profileLoopStart("filtfilt_loop_6", __LINE__, (i1 - 1) + 1, "1593");
      for (int32_T ii{0}; ii < i1; ii++) {
        int32_T b_i1;
        int32_T b_loop_ub;
        int32_T i7;
        int32_T i9;
        int32_T loop_ub;
        real32_T f1;
        i = r->size[0] * r->size[1];
        r->size[0] = 1;
        r->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(r, i, &e_emlrtRTEI);
        i = static_cast<int32_T>(Letr + 1.0F) - 1;
        b_i1 = xc1->size[0] * xc1->size[1];
        xc1->size[0] = static_cast<int32_T>(Letr + 1.0F) - 1;
        xc1->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(xc1, b_i1, &i_emlrtRTEI);
        profileLoopStart("filtfilt_loop_10", __LINE__,
                         (cpu_xc1->size[1] - 1) + 1, "1594");
        for (b_i1 = 0; b_i1 < cpu_xc1->size[1]; b_i1++) {
          r->data[b_i1] = 2.0F * cpu_xc1->data[cpu_xc1->size[0] * b_i1];
          profileLoopStart("filtfilt_loop_13", __LINE__, (i - 1) + 1, "1595");
          for (loop_ub = 0; loop_ub < i; loop_ub++) {
            xc1->data[loop_ub + xc1->size[0] * b_i1] =
                cpu_xc1->data[((static_cast<int32_T>(Letr + 1.0F) - loop_ub) +
                               cpu_xc1->size[0] * b_i1) -
                              1];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        if (xt_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_xt, &c_gpu_xt);
        }
        nvtxMarkA("#bsxfun#" MW_AT_LINE);
        bsxfun(r, xc1, cpu_xt);
        xt_outdatedOnCpu = false;
        xt_outdatedOnGpu = true;
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = cpu_B->size[0] * cpu_B->size[1];
        cpu_B->size[0] = 1;
        cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_B, i, &l_emlrtRTEI);
        profileLoopStart("filtfilt_loop_17", __LINE__, (B->size[1] - 1) + 1,
                         "1596");
        for (i = 0; i < B->size[1]; i++) {
          cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = cpu_A->size[0] * cpu_A->size[1];
        cpu_A->size[0] = 1;
        cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_A, i, &n_emlrtRTEI);
        profileLoopStart("filtfilt_loop_20", __LINE__, (A->size[1] - 1) + 1,
                         "1597");
        for (i = 0; i < A->size[1]; i++) {
          cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        zi_outdatedOnCpu = false;
        zi_outdatedOnGpu = false;
        i = cpu_zi->size[0] * cpu_zi->size[1];
        cpu_zi->size[0] = zi->size[0];
        cpu_zi->size[1] = cpu_xt->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_zi, i, &q_emlrtRTEI);
        b_loop_ub = cpu_xt->size[1];
        profileLoopStart("filtfilt_loop_23", __LINE__, (b_loop_ub - 1) + 1,
                         "1598");
        for (i = 0; i < b_loop_ub; i++) {
          loop_ub = zi->size[0];
          profileLoopStart("filtfilt_loop_25", __LINE__, (loop_ub - 1) + 1,
                           "1599");
          for (b_i1 = 0; b_i1 < loop_ub; b_i1++) {
            cpu_zi->data[b_i1 + cpu_zi->size[0] * i] =
                zi->data[b_i1 + zi->size[0] * ii] *
                cpu_xt->data[cpu_xt->size[0] * i];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        nvtxMarkA("#c_filter#" MW_AT_LINE);
        c_filter(cpu_B, &B_outdatedOnCpu, &h_gpu_B, &B_outdatedOnGpu, cpu_A,
                 &A_outdatedOnCpu, &h_gpu_A, &A_outdatedOnGpu, cpu_xt,
                 &c_gpu_xt, &xt_outdatedOnGpu, cpu_zi, &zi_outdatedOnCpu,
                 &e_gpu_zi, &zi_outdatedOnGpu, cpu__3, &yc5_outdatedOnGpu,
                 &gpu__3, &yc5_outdatedOnCpu, cpu_zo, &zo_outdatedOnCpu,
                 &d_gpu_zo, &zo_outdatedOnGpu);
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = b_cpu_B->size[0] * b_cpu_B->size[1];
        b_cpu_B->size[0] = 1;
        b_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_cpu_B, i, &t_emlrtRTEI);
        profileLoopStart("filtfilt_loop_28", __LINE__, (B->size[1] - 1) + 1,
                         "1600");
        for (i = 0; i < B->size[1]; i++) {
          b_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = b_cpu_A->size[0] * b_cpu_A->size[1];
        b_cpu_A->size[0] = 1;
        b_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_cpu_A, i, &u_emlrtRTEI);
        profileLoopStart("filtfilt_loop_31", __LINE__, (A->size[1] - 1) + 1,
                         "1601");
        for (i = 0; i < A->size[1]; i++) {
          b_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        nvtxMarkA("#c_filter#" MW_AT_LINE);
        c_filter(b_cpu_B, &B_outdatedOnCpu, &i_gpu_B, &B_outdatedOnGpu, b_cpu_A,
                 &A_outdatedOnCpu, &i_gpu_A, &A_outdatedOnGpu, cpu_xc1,
                 &gpu_xc1, &yc3_outdatedOnGpu, cpu_zo, &zo_outdatedOnCpu,
                 &d_gpu_zo, &zo_outdatedOnGpu, cpu_yc2, &yc2_outdatedOnCpu,
                 &c_gpu_yc2, &a__2_outdatedOnGpu, c_cpu_zo, &b_zo_outdatedOnCpu,
                 &e_gpu_zo, &b_zo_outdatedOnGpu);
        f1 = static_cast<real32_T>(cpu_xc1->size[0]) - Letr;
        if (f1 > static_cast<real32_T>(static_cast<real_T>(cpu_xc1->size[0]) -
                                       1.0)) {
          loop_ub = 0;
          i7 = 1;
          i9 = -1;
        } else {
          loop_ub = static_cast<int32_T>(static_cast<real32_T>(
                        static_cast<real_T>(cpu_xc1->size[0]) - 1.0)) -
                    1;
          i7 = -1;
          i9 = static_cast<int32_T>(f1) - 1;
        }
        i = r1->size[0] * r1->size[1];
        r1->size[0] = 1;
        r1->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(r1, i, &e_emlrtRTEI);
        i = b_xc1->size[0] * b_xc1->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        b_xc1->size[0] = div_s32(i9 - loop_ub, i7) + 1;
        b_xc1->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_xc1, i, &cb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_36", __LINE__,
                         (cpu_xc1->size[1] - 1) + 1, "1602");
        for (i = 0; i < cpu_xc1->size[1]; i++) {
          if (a__4_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_xc1, &gpu_xc1);
          }
          a__4_outdatedOnGpu = false;
          r1->data[i] =
              2.0F *
              cpu_xc1->data[(cpu_xc1->size[0] + cpu_xc1->size[0] * i) - 1];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_38", __LINE__,
                           div_s32(i9 - loop_ub, i7) + 1, "1603");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i9 - loop_ub, i7); b_i1++) {
            b_xc1->data[b_i1 + b_xc1->size[0] * i] =
                cpu_xc1->data[(loop_ub + i7 * b_i1) + cpu_xc1->size[0] * i];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = g_cpu_B->size[0] * g_cpu_B->size[1];
        g_cpu_B->size[0] = 1;
        g_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(g_cpu_B, i, &eb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_40", __LINE__, (B->size[1] - 1) + 1,
                         "1604");
        for (i = 0; i < B->size[1]; i++) {
          g_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = g_cpu_A->size[0] * g_cpu_A->size[1];
        g_cpu_A->size[0] = 1;
        g_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(g_cpu_A, i, &fb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_43", __LINE__, (A->size[1] - 1) + 1,
                         "1605");
        for (i = 0; i < A->size[1]; i++) {
          g_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        if (r2_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_r2, &gpu_r2);
        }
        nvtxMarkA("#bsxfun#" MW_AT_LINE);
        bsxfun(r1, b_xc1, cpu_r2);
        r2_outdatedOnCpu = false;
        a__2_outdatedOnGpu = true;
        nvtxMarkA("#d_filter#" MW_AT_LINE);
        d_filter(g_cpu_B, &B_outdatedOnCpu, &j_gpu_B, &B_outdatedOnGpu, g_cpu_A,
                 &A_outdatedOnCpu, &j_gpu_A, &A_outdatedOnGpu, cpu_r2, &gpu_r2,
                 &a__2_outdatedOnGpu, c_cpu_zo, &b_zo_outdatedOnCpu, &e_gpu_zo,
                 &b_zo_outdatedOnGpu, cpu_yc3, &yc5_outdatedOnGpu, &c_gpu_yc3,
                 &yc3_outdatedOnGpu);
        if (cpu_yc3->size[0] < 1) {
          loop_ub = 0;
          i7 = 1;
          i9 = -1;
        } else {
          loop_ub = cpu_yc3->size[0] - 1;
          i7 = -1;
          i9 = 0;
        }
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = i_cpu_B->size[0] * i_cpu_B->size[1];
        i_cpu_B->size[0] = 1;
        i_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(i_cpu_B, i, &ib_emlrtRTEI);
        profileLoopStart("filtfilt_loop_47", __LINE__, (B->size[1] - 1) + 1,
                         "1606");
        for (i = 0; i < B->size[1]; i++) {
          i_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = i_cpu_A->size[0] * i_cpu_A->size[1];
        i_cpu_A->size[0] = 1;
        i_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(i_cpu_A, i, &kb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_49", __LINE__, (A->size[1] - 1) + 1,
                         "1607");
        for (i = 0; i < A->size[1]; i++) {
          i_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        yc5_outdatedOnCpu = false;
        yc3_outdatedOnGpu = false;
        i = c_cpu_yc3->size[0] * c_cpu_yc3->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        c_cpu_yc3->size[0] = div_s32(i9 - loop_ub, i7) + 1;
        c_cpu_yc3->size[1] = cpu_yc3->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(c_cpu_yc3, i, &mb_emlrtRTEI);
        zi_outdatedOnCpu = false;
        zi_outdatedOnGpu = false;
        i = e_cpu_zi->size[0] * e_cpu_zi->size[1];
        e_cpu_zi->size[0] = zi->size[0];
        e_cpu_zi->size[1] = cpu_yc3->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(e_cpu_zi, i, &q_emlrtRTEI);
        profileLoopStart("filtfilt_loop_51", __LINE__,
                         (cpu_yc3->size[1] - 1) + 1, "1608");
        for (i = 0; i < cpu_yc3->size[1]; i++) {
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_53", __LINE__,
                           div_s32(i9 - loop_ub, i7) + 1, "1609");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i9 - loop_ub, i7); b_i1++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_yc3, &c_gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            c_cpu_yc3->data[b_i1 + c_cpu_yc3->size[0] * i] =
                cpu_yc3->data[(loop_ub + i7 * b_i1) + cpu_yc3->size[0] * i];
            yc3_outdatedOnGpu = true;
          }
          profileLoopEnd();
          b_loop_ub = zi->size[0];
          profileLoopStart("filtfilt_loop_54", __LINE__, (b_loop_ub - 1) + 1,
                           "1610");
          for (b_i1 = 0; b_i1 < b_loop_ub; b_i1++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_yc3, &c_gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            e_cpu_zi->data[b_i1 + e_cpu_zi->size[0] * i] =
                zi->data[b_i1 + zi->size[0] * ii] *
                cpu_yc3->data[(cpu_yc3->size[0] + cpu_yc3->size[0] * i) - 1];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        nvtxMarkA("#c_filter#" MW_AT_LINE);
        c_filter(i_cpu_B, &B_outdatedOnCpu, &k_gpu_B, &B_outdatedOnGpu, i_cpu_A,
                 &A_outdatedOnCpu, &k_gpu_A, &A_outdatedOnGpu, c_cpu_yc3,
                 &d_gpu_yc3, &yc3_outdatedOnGpu, e_cpu_zi, &zi_outdatedOnCpu,
                 &f_gpu_zi, &zi_outdatedOnGpu, cpu__4, &yc5_outdatedOnGpu,
                 &gpu__4, &a__4_outdatedOnGpu, e_cpu_zo, &b_zo_outdatedOnCpu,
                 &f_gpu_zo, &b_zo_outdatedOnGpu);
        if (cpu_yc2->size[0] < 1) {
          loop_ub = 0;
          i7 = 1;
          i9 = -1;
        } else {
          loop_ub = cpu_yc2->size[0] - 1;
          i7 = -1;
          i9 = 0;
        }
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = k_cpu_B->size[0] * k_cpu_B->size[1];
        k_cpu_B->size[0] = 1;
        k_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(k_cpu_B, i, &ob_emlrtRTEI);
        profileLoopStart("filtfilt_loop_55", __LINE__, (B->size[1] - 1) + 1,
                         "1611");
        for (i = 0; i < B->size[1]; i++) {
          k_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = k_cpu_A->size[0] * k_cpu_A->size[1];
        k_cpu_A->size[0] = 1;
        k_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(k_cpu_A, i, &qb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_57", __LINE__, (A->size[1] - 1) + 1,
                         "1612");
        for (i = 0; i < A->size[1]; i++) {
          k_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        a__2_outdatedOnGpu = false;
        i = c_cpu_yc2->size[0] * c_cpu_yc2->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        c_cpu_yc2->size[0] = div_s32(i9 - loop_ub, i7) + 1;
        c_cpu_yc2->size[1] = cpu_yc2->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(c_cpu_yc2, i, &rb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_60", __LINE__,
                         (cpu_yc2->size[1] - 1) + 1, "1613");
        for (i = 0; i < cpu_yc2->size[1]; i++) {
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_61", __LINE__,
                           div_s32(i9 - loop_ub, i7) + 1, "1614");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i9 - loop_ub, i7); b_i1++) {
            if (yc2_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_yc2, &c_gpu_yc2);
            }
            yc2_outdatedOnCpu = false;
            c_cpu_yc2->data[b_i1 + c_cpu_yc2->size[0] * i] =
                cpu_yc2->data[(loop_ub + i7 * b_i1) + cpu_yc2->size[0] * i];
            a__2_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        nvtxMarkA("#d_filter#" MW_AT_LINE);
        d_filter(k_cpu_B, &B_outdatedOnCpu, &l_gpu_B, &B_outdatedOnGpu, k_cpu_A,
                 &A_outdatedOnCpu, &l_gpu_A, &A_outdatedOnGpu, c_cpu_yc2,
                 &d_gpu_yc2, &a__2_outdatedOnGpu, e_cpu_zo, &b_zo_outdatedOnCpu,
                 &f_gpu_zo, &b_zo_outdatedOnGpu, cpu_yc5, &yc5_outdatedOnCpu,
                 &b_gpu_yc5, &yc5_outdatedOnGpu);
        if (cpu_yc5->size[0] < 1) {
          loop_ub = 0;
          i7 = 1;
          i9 = -1;
        } else {
          loop_ub = cpu_yc5->size[0] - 1;
          i7 = -1;
          i9 = 0;
        }
        a__4_outdatedOnGpu = false;
        yc3_outdatedOnGpu = false;
        i = cpu_xc1->size[0] * cpu_xc1->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        cpu_xc1->size[0] = div_s32(i9 - loop_ub, i7) + 1;
        cpu_xc1->size[1] = cpu_yc5->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_xc1, i, &tb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_63", __LINE__,
                         (cpu_yc5->size[1] - 1) + 1, "1615");
        for (i = 0; i < cpu_yc5->size[1]; i++) {
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_64", __LINE__,
                           div_s32(i9 - loop_ub, i7) + 1, "1616");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i9 - loop_ub, i7); b_i1++) {
            if (yc5_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_yc5, &b_gpu_yc5);
            }
            yc5_outdatedOnCpu = false;
            cpu_xc1->data[b_i1 + cpu_xc1->size[0] * i] =
                cpu_yc5->data[(loop_ub + i7 * b_i1) + cpu_yc5->size[0] * i];
            yc3_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
      }
      profileLoopEnd();
      i = y->size[0] * y->size[1] * y->size[2];
      y->size[0] = X->size[0];
      y->size[1] = X->size[1];
      y->size[2] = X->size[2];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(y, i, &f_emlrtRTEI);
      profileLoopStart(
          "filtfilt_loop_9", __LINE__,
          (static_cast<int32_T>(sz_idx_0) * static_cast<int32_T>(sz_idx_1) *
               static_cast<int32_T>(sz_idx_2) -
           1) +
              1,
          "1617");
      for (i = 0;
           i < static_cast<int32_T>(sz_idx_0) * static_cast<int32_T>(sz_idx_1) *
                   static_cast<int32_T>(sz_idx_2);
           i++) {
        y->data[i] = cpu_xc1->data[i];
      }
      profileLoopEnd();
    }
    if (xIsRow) {
      i = y->size[0] * y->size[1] * y->size[2];
      y->size[0] = 1;
      y->size[1] = X->size[0];
      y->size[2] = 1;
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(y, i, &g_emlrtRTEI);
    }
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_ytemp);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_r2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_yc2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&l_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&l_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_yc2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&k_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&k_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f_cpu_zi);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_yc3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&j_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&j_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&e_cpu_zi);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_yc3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&i_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&i_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_xt);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&h_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&h_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&g_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&g_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_zi);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_xc1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&r1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&ytemp);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_zi);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&e_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&e_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_zi);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_xt);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_zi);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&xc1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&r);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_yc5);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_yc5);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f_cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu__2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&e_cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu__4);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_yc3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_yc3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_ytemp);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_yc2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_yc2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu__1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_zo);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu__3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&xt);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_xt);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_xc1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_ytemp);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&zi);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&B);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&X);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&den);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&num);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_ytemp);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_xc1);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_xt);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu__3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu__1);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&e_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_ytemp);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu__4);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&f_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu__2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_yc5);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_yc5);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&h_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&h_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&e_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&i_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&i_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_xt);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_zi);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&j_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&j_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&e_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&e_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_xt);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&k_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&k_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&f_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&f_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&f_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&l_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&l_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&g_gpu_B);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&g_gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_r2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_ytemp);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

} // namespace coder

// End of code generation (filtfilt.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filtfilt_mexutil.cu
//
// Code generation for function 'filtfilt_mexutil'
//

// Include files
#include "filtfilt_mexutil.h"
#include "filtfilt_data.h"
#include "filtfilt_types.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cstring>

// Function Definitions
void checkCudaError(hipError_t errorCode, const char_T *file, int32_T b_line)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errorCode != hipSuccess) {
    nvtxMarkA("#gpuThrowError#" MW_AT_LINE);
    gpuThrowError(errorCode, hipGetErrorName(errorCode),
                  hipGetErrorString(errorCode), file, b_line);
  }
  nvtxRangePop();
}

int64_T computeEndIdx(int64_T start, int64_T end, int64_T stride)
{
  int64_T newEnd;
  nvtxRangePushA("#fcn#computeEndIdx#" MW_AT_LOCATION);
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  nvtxRangePop();
  return newEnd;
}

void gpuEmxFree_real32_T(emxArray_real32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real32_T#" MW_AT_LOCATION);
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                   emxArray_real32_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyGpuToCpu_real32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(cpu->data, gpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

void gpuEmxReset_real32_T(emxArray_real32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real32_T));
  nvtxRangePop();
}

void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                   const char_T *errorString, const char_T *file,
                   int32_T b_line)
{
  emlrtRTEInfo rtInfo;
  nvtxRangePushA("#fcn#gpuThrowError#" MW_AT_LOCATION);
  rtInfo.lineNo = b_line;
  rtInfo.colNo = 0;
  rtInfo.fName = "";
  rtInfo.pName = file;
  emlrtCUDAError(errorCode, (char_T *)errorName, (char_T *)errorString, &rtInfo,
                 emlrtRootTLSGlobal);
  nvtxRangePop();
}

// End of code generation (filtfilt_mexutil.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filtfilt_terminate.cu
//
// Code generation for function 'filtfilt_terminate'
//

// Include files
#include "filtfilt_terminate.h"
#include "_coder_filtfilt_mex.h"
#include "filtfilt_data.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void checkCudaError(hipError_t errorCode);

static void emlrtExitTimeCleanupDtorFcn(const void *r);

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString);

// Function Definitions
static void checkCudaError(hipError_t errorCode)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errorCode != hipSuccess) {
    nvtxMarkA("#gpuThrowError#" MW_AT_LINE);
    gpuThrowError(errorCode, hipGetErrorName(errorCode),
                  hipGetErrorString(errorCode));
  }
  nvtxRangePop();
}

static void emlrtExitTimeCleanupDtorFcn(const void *r)
{
  nvtxRangePushA("#fcn#emlrtExitTimeCleanupDtorFcn#" MW_AT_LOCATION);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  nvtxRangePop();
}

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString)
{
  nvtxRangePushA("#fcn#gpuThrowError#" MW_AT_LOCATION);
  emlrtThinCUDAError(errorCode, (char_T *)errorName, (char_T *)errorString,
                     (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  nvtxRangePop();
}

void filtfilt_atexit()
{
  nvtxRangePushA("#termFcn#filtfilt_atexit#" MW_AT_LOCATION);
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  try {
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&emlrtExitTimeCleanupDtorFcn,
                                      nullptr, nullptr, nullptr);
    emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
    emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
    emlrtExitTimeCleanup(&emlrtContextGlobal);
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  nvtxRangePop();
}

void filtfilt_terminate()
{
  nvtxRangePushA("#fcn#filtfilt_terminate#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  checkCudaError(hipGetLastError());
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  nvtxRangePop();
}

// End of code generation (filtfilt_terminate.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filter.cu
//
// Code generation for function 'filter'
//

// Include files
#include "filter.h"
#include "ec_filtfilt_fp_data.h"
#include "ec_filtfilt_fp_emxutil.h"
#include "ec_filtfilt_fp_mexutil.h"
#include "ec_filtfilt_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo l_emlrtRTEI{
    1,               // lineNo
    1,               // colNo
    "gpufilterImpl", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
    "gpufilterImpl.p" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "stencil_codegen", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "stencil_codegen.p" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    172,                                                          // lineNo
    20,                                                           // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    164,                                                               // lineNo
    9,                                                                 // colNo
    "filter",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/filter.m" // pName
};

// Function Declarations
static __global__ void filter_kernel20(const emxArray_real32_T a, real32_T *a1);

static __global__ void filter_kernel21(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel22(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T a);

static __global__ void filter_kernel23(emxArray_real32_T a);

static __global__ void filter_kernel24(const int32_T k, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel25(const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void filter_kernel26(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void filter_kernel27(const emxArray_real32_T x,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void
filter_kernel28(const emxArray_real32_T expanded, const emxArray_int32_T rows,
                const emxArray_real32_T b, const int32_T c,
                emxArray_real32_T convOut, int32_T b_dim0);

static __global__ void filter_kernel29(const emxArray_real32_T convOut,
                                       const int32_T na, const int32_T b,
                                       emxArray_real32_T zf);

static __global__ void filter_kernel30(const emxArray_real32_T convOut,
                                       const emxArray_real32_T a,
                                       const int32_T b, emxArray_real32_T zf,
                                       int32_T a_dim0, int32_T x_dim0);

static __global__ void filter_kernel31(const emxArray_real32_T convOut,
                                       const int32_T x, emxArray_real32_T y);

static __global__ void filter_kernel32(const emxArray_real32_T zi,
                                       const int32_T b,
                                       emxArray_real32_T convOut);

static __global__ void filter_kernel33(const emxArray_real32_T a, real32_T *a1);

static __global__ void filter_kernel34(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel35(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T a);

static __global__ void filter_kernel36(emxArray_real32_T a);

static __global__ void filter_kernel37(const int32_T k, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel38(const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void filter_kernel39(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void filter_kernel40(const emxArray_real32_T x,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void
filter_kernel41(const emxArray_real32_T expanded, const emxArray_int32_T rows,
                const emxArray_real32_T b, const int32_T c,
                emxArray_real32_T convOut, int32_T b_dim0);

static __global__ void filter_kernel42(const emxArray_real32_T convOut,
                                       const int32_T x, emxArray_real32_T y);

static __global__ void filter_kernel43(const emxArray_real32_T zi,
                                       const int32_T b,
                                       emxArray_real32_T convOut);

static void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu,
                                         boolean_T needsCopy);

static void gpuEmxFree_int32_T(emxArray_int32_T *gpu);

static void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                         const emxArray_int32_T *cpu);

static void gpuEmxReset_int32_T(emxArray_int32_T *gpu);

// Function Definitions
static __global__
    __launch_bounds__(32, 1) void filter_kernel20(const emxArray_real32_T a,
                                                  real32_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel21(
    const real32_T *a1, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel22(const real32_T *a1,
                                                            const int32_T b,
                                                            emxArray_real32_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel23(emxArray_real32_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel24(
    const int32_T k, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    b_k = static_cast<int32_T>(idx);
    b_b.data[b_k + k] = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void filter_kernel25(const int32_T b,
                                                    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[k] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel26(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    int32_T k;
    k = static_cast<int32_T>(idx);
    j = y.data[k];
    if ((offsetH < 0) && (j < MIN_int32_T - offsetH)) {
      j = MIN_int32_T;
    } else if ((offsetH > 0) && (j > MAX_int32_T - offsetH)) {
      j = MAX_int32_T;
    } else {
      j += offsetH;
    }
    iv.data[k] = j - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel27(
    const emxArray_real32_T x, const emxArray_int32_T iv, const int32_T b,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[iv.data[k]] = x.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel28(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_real32_T b, const int32_T c, emxArray_real32_T convOut,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    real32_T cv;
    k = static_cast<int32_T>(idx);
    cv = 0.0F;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T j;
      j = rows.data[m];
      if (j > 2147483646 - k) {
        j = MAX_int32_T;
      } else {
        j = (k + j) + 1;
      }
      cv += expanded.data[j - 1] * b.data[(b_dim0 - m) - 1];
    }
    convOut.data[k] = cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel29(
    const emxArray_real32_T convOut, const int32_T na, const int32_T b,
    emxArray_real32_T zf)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    zf.data[k] = convOut.data[na + k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel30(
    const emxArray_real32_T convOut, const emxArray_real32_T a, const int32_T b,
    emxArray_real32_T zf, int32_T a_dim0, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    i = static_cast<int32_T>(idx);
    k = static_cast<int32_T>(
        fmin(static_cast<real_T>(i) + 2.0, static_cast<real_T>(a_dim0)));
    for (int32_T j{0}; j <= k - 2; j++) {
      zf.data[i + 1] += convOut.data[(x_dim0 + i) - j] * a.data[j + 1];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel31(
    const emxArray_real32_T convOut, const int32_T x, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = convOut.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel32(
    const emxArray_real32_T zi, const int32_T b, emxArray_real32_T convOut)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    convOut.data[i] += zi.data[i];
  }
}

static __global__
    __launch_bounds__(32, 1) void filter_kernel33(const emxArray_real32_T a,
                                                  real32_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel34(
    const real32_T *a1, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel35(const real32_T *a1,
                                                            const int32_T b,
                                                            emxArray_real32_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel36(emxArray_real32_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel37(
    const int32_T k, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    b_k = static_cast<int32_T>(idx);
    b_b.data[b_k + k] = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void filter_kernel38(const int32_T b,
                                                    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[k] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel39(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int32_T q1;
    k = static_cast<int32_T>(idx);
    q1 = y.data[k];
    if ((offsetH < 0) && (q1 < MIN_int32_T - offsetH)) {
      q1 = MIN_int32_T;
    } else if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[k] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel40(
    const emxArray_real32_T x, const emxArray_int32_T iv, const int32_T b,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[iv.data[k]] = x.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel41(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_real32_T b, const int32_T c, emxArray_real32_T convOut,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    real32_T cv;
    k = static_cast<int32_T>(idx);
    cv = 0.0F;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - k) {
        q1 = MAX_int32_T;
      } else {
        q1 = (k + q1) + 1;
      }
      cv += expanded.data[q1 - 1] * b.data[(b_dim0 - m) - 1];
    }
    convOut.data[k] = cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel42(
    const emxArray_real32_T convOut, const int32_T x, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = convOut.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel43(
    const emxArray_real32_T zi, const int32_T b, emxArray_real32_T convOut)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    convOut.data[k] += zi.data[k];
  }
}

static void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu,
                                         boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  int32_T *newData;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_int32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(int32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(int32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

static void gpuEmxFree_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_int32_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

static void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                         const emxArray_int32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(int32_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

static void gpuEmxReset_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_int32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_int32_T));
  nvtxRangePop();
}

//
//
namespace coder {
void b_filter(emxArray_real32_T *cpu_b, boolean_T *b_outdatedOnCpu,
              emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
              emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
              emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
              emxArray_real32_T *cpu_x, emxArray_real32_T *gpu_x,
              boolean_T *x_outdatedOnGpu, emxArray_real32_T *cpu_zi,
              boolean_T *zi_outdatedOnCpu, emxArray_real32_T *gpu_zi,
              boolean_T *zi_outdatedOnGpu, emxArray_real32_T *cpu_y,
              boolean_T *y_outdatedOnCpu, emxArray_real32_T *gpu_y,
              boolean_T *y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_real32_T gpu_convOut;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *cpu_convOut;
  emxArray_real32_T *cpu_expanded;
  int32_T k;
  int32_T na;
  int32_T nb;
  real32_T *gpu_a1;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#b_filter#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_expanded);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a1, 4UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[0];
  nb = cpu_b->size[0];
  nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
  }
  nvtxMarkA("#filter_kernel33#" MW_AT_LINE);
  filter_kernel33<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  if (*a_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0])) &&
      (!(cpu_a->data[0] == 0.0F)) && (cpu_a->data[0] != 1.0F)) {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel34#" MW_AT_LINE);
      filter_kernel34<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel35#" MW_AT_LINE);
      filter_kernel35<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
    }
    nvtxMarkA("#filter_kernel36#" MW_AT_LINE);
    filter_kernel36<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
    *a_outdatedOnCpu = true;
  }
  if (cpu_a->size[0] > cpu_b->size[0]) {
    nb = cpu_a->size[0] - cpu_b->size[0];
    k = cpu_b->size[0];
    na = cpu_b->size[0];
    cpu_b->size[0] += nb;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_b, na, &l_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel37#" MW_AT_LINE);
      filter_kernel37<<<grid, block>>>(k, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
  }
  if (cpu_x->size[0] == 0) {
    *y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
  } else {
    real_T blockDims_idx_0;
    int32_T KH;
    int32_T offsetH;
    uint32_T OH;
    uint32_T window_idx_0;
    int8_T threadDims_idx_0;
    boolean_T convOut_outdatedOnCpu;
    boolean_T convOut_outdatedOnGpu;
    window_idx_0 = static_cast<uint32_T>(cpu_b->size[0]);
    OH = (static_cast<uint32_T>(cpu_x->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    na = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    blockDims_idx_0 =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(na) - 1.0)) /
                   static_cast<real_T>(na));
    threadDims_idx_0 = static_cast<int8_T>(na);
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    KH = cpu_b->size[0];
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&cpu_expanded, 1, &m_emlrtRTEI, true);
    if ((static_cast<int32_T>(OH) > 0) &&
        (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH))) {
      na = MAX_int32_T;
    } else {
      na = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0];
    cpu_expanded->size[0] = na - 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_expanded, k, &m_emlrtRTEI);
    na = (static_cast<int32_T>(OH) + static_cast<int32_T>(window_idx_0)) - 2;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel38#" MW_AT_LINE);
      filter_kernel38<<<grid, block>>>(na, gpu_expanded);
    }
    nb = cpu_x->size[0];
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_cpu_y, 2, &m_emlrtRTEI, true);
    k = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(b_cpu_y, k, &n_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    na = 1;
    profileLoopStart("b_filter_loop_0", __LINE__, (nb - 2) + 1, "");
    for (k = 0; k <= nb - 2; k++) {
      na++;
      b_cpu_y->data[k + 1] = na;
    }
    profileLoopEnd();
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&cpu_iv, 1, &m_emlrtRTEI, true);
    k = cpu_iv->size[0];
    cpu_iv->size[0] = b_cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_iv, k, &m_emlrtRTEI);
    na = b_cpu_y->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel39#" MW_AT_LINE);
      filter_kernel39<<<grid, block>>>(offsetH, b_gpu_y, na, gpu_iv);
    }
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_cpu_y);
    nb = cpu_iv->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    *x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel40#" MW_AT_LINE);
      filter_kernel40<<<grid, block>>>(*gpu_x, gpu_iv, nb - 1, gpu_expanded);
    }
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&cpu_iv);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&cpu_rows, 2, &m_emlrtRTEI, true);
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = static_cast<int32_T>(window_idx_0);
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_rows, k, &n_emlrtRTEI);
    cpu_rows->data[0] = 0;
    na = 0;
    profileLoopStart("b_filter_loop_1", __LINE__, (KH - 2) + 1, "");
    for (k = 0; k <= KH - 2; k++) {
      na++;
      cpu_rows->data[k + 1] = na;
    }
    profileLoopEnd();
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&cpu_convOut, 1, &l_emlrtRTEI, true);
    k = cpu_convOut->size[0];
    cpu_convOut->size[0] = static_cast<int32_T>(OH);
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_convOut, k, &m_emlrtRTEI);
    if (blockDims_idx_0 < 4.294967296E+9) {
      if (blockDims_idx_0 >= 0.0) {
        window_idx_0 = static_cast<uint32_T>(blockDims_idx_0);
      } else {
        window_idx_0 = 0U;
      }
    } else if (blockDims_idx_0 >= 4.294967296E+9) {
      window_idx_0 = MAX_uint32_T;
    } else {
      window_idx_0 = 0U;
    }
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwApplyLaunchParameters(
        computeNumIters(static_cast<int32_T>(OH) - 1),
        dim3(window_idx_0, 1U, 1U),
        dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid, &block);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel41#" MW_AT_LINE);
      filter_kernel41<<<grid, block>>>(gpu_expanded, gpu_rows, *gpu_b,
                                       static_cast<int32_T>(OH) - 1,
                                       gpu_convOut, cpu_b->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    convOut_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&cpu_rows);
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&cpu_expanded);
    if (cpu_zi->size[0] != 0) {
      if (cpu_zi->size[0] == 1) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        cpu_convOut->data[0] += cpu_zi->data[0];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      } else {
        na = cpu_zi->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel43#" MW_AT_LINE);
          filter_kernel43<<<grid, block>>>(*gpu_zi, na - 1, gpu_convOut);
        }
      }
    }
    na = cpu_x->size[0] + cpu_a->size[0];
    profileLoopStart("b_filter_loop_2", __LINE__, (na - 2) + 1, "");
    for (k = 0; k <= na - 2; k++) {
      nb = static_cast<int32_T>(std::fmin(static_cast<real_T>(k) + 1.0,
                                          static_cast<real_T>(cpu_a->size[0])));
      profileLoopStart("b_filter_loop_3", __LINE__, (nb - 2) + 1, "");
      for (offsetH = 0; offsetH <= nb - 2; offsetH++) {
        if (convOut_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        }
        if (*a_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
        }
        *a_outdatedOnCpu = false;
        cpu_convOut->data[k] -=
            cpu_convOut->data[(k - offsetH) - 1] * cpu_a->data[offsetH + 1];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    k = cpu_y->size[0];
    cpu_y->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_y, k, &o_emlrtRTEI);
    na = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut,
                                    !convOut_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_y, gpu_y, true);
    if (convOut_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel42#" MW_AT_LINE);
      filter_kernel42<<<grid, block>>>(gpu_convOut, na, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&cpu_convOut);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_convOut);
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
  nvtxRangePop();
}

//
//
void filter(emxArray_real32_T *cpu_b, boolean_T *b_outdatedOnCpu,
            emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
            emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
            emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
            emxArray_real32_T *cpu_x, emxArray_real32_T *gpu_x,
            boolean_T *x_outdatedOnGpu, emxArray_real32_T *cpu_zi,
            boolean_T *zi_outdatedOnCpu, emxArray_real32_T *gpu_zi,
            boolean_T *zi_outdatedOnGpu, emxArray_real32_T *cpu_y,
            boolean_T *y_outdatedOnCpu, emxArray_real32_T *gpu_y,
            boolean_T *y_outdatedOnGpu, emxArray_real32_T *cpu_zf,
            boolean_T *zf_outdatedOnCpu, emxArray_real32_T *gpu_zf,
            boolean_T *zf_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_real32_T gpu_convOut;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *cpu_convOut;
  emxArray_real32_T *cpu_expanded;
  int32_T k;
  int32_T na;
  int32_T nb;
  int32_T zfSize_idx_0;
  real32_T *gpu_a1;
  boolean_T a_needsGpuEnsureCapacity;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#filter#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_expanded);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a1, 4UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[0];
  nb = cpu_b->size[0];
  nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
  }
  nvtxMarkA("#filter_kernel20#" MW_AT_LINE);
  filter_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  if (*a_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0]))) {
    if (!(cpu_a->data[0] == 0.0F)) {
      if (cpu_a->data[0] != 1.0F) {
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
        b_needsGpuEnsureCapacity = false;
        if (*b_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel21#" MW_AT_LINE);
          filter_kernel21<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
        }
        *b_outdatedOnGpu = false;
        *b_outdatedOnCpu = true;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel22#" MW_AT_LINE);
          filter_kernel22<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
        }
        a_needsGpuEnsureCapacity = false;
        nvtxMarkA("#filter_kernel23#" MW_AT_LINE);
        filter_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
        *a_outdatedOnCpu = true;
      } else {
        a_needsGpuEnsureCapacity = true;
      }
    } else {
      a_needsGpuEnsureCapacity = true;
    }
  } else {
    a_needsGpuEnsureCapacity = true;
  }
  if (cpu_a->size[0] > cpu_b->size[0]) {
    nb = cpu_a->size[0] - cpu_b->size[0];
    k = cpu_b->size[0];
    na = cpu_b->size[0];
    cpu_b->size[0] += nb;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_b, na, &l_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel24#" MW_AT_LINE);
      filter_kernel24<<<grid, block>>>(k, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
  }
  zfSize_idx_0 = cpu_b->size[0] - 1;
  if (cpu_x->size[0] == 0) {
    *y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
    *zf_outdatedOnCpu = false;
    k = cpu_zf->size[0];
    cpu_zf->size[0] = cpu_b->size[0] - 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_zf, k, &l_emlrtRTEI);
  } else {
    real_T blockDims_idx_0;
    int32_T KH;
    int32_T offsetH;
    uint32_T OH;
    uint32_T window_idx_0;
    int8_T threadDims_idx_0;
    boolean_T convOut_outdatedOnCpu;
    boolean_T convOut_outdatedOnGpu;
    window_idx_0 = static_cast<uint32_T>(cpu_b->size[0]);
    OH = (static_cast<uint32_T>(cpu_x->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    na = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    blockDims_idx_0 =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(na) - 1.0)) /
                   static_cast<real_T>(na));
    threadDims_idx_0 = static_cast<int8_T>(na);
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    KH = cpu_b->size[0];
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&cpu_expanded, 1, &m_emlrtRTEI, true);
    if ((static_cast<int32_T>(OH) > 0) &&
        (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH))) {
      na = MAX_int32_T;
    } else {
      na = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0];
    cpu_expanded->size[0] = na - 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_expanded, k, &m_emlrtRTEI);
    nb = (static_cast<int32_T>(OH) + static_cast<int32_T>(window_idx_0)) - 2;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel25#" MW_AT_LINE);
      filter_kernel25<<<grid, block>>>(nb, gpu_expanded);
    }
    nb = cpu_x->size[0];
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_cpu_y, 2, &m_emlrtRTEI, true);
    k = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(b_cpu_y, k, &n_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    na = 1;
    profileLoopStart("filter_loop_0", __LINE__, (nb - 2) + 1, "");
    for (k = 0; k <= nb - 2; k++) {
      na++;
      b_cpu_y->data[k + 1] = na;
    }
    profileLoopEnd();
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&cpu_iv, 1, &m_emlrtRTEI, true);
    k = cpu_iv->size[0];
    cpu_iv->size[0] = b_cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_iv, k, &m_emlrtRTEI);
    nb = b_cpu_y->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel26#" MW_AT_LINE);
      filter_kernel26<<<grid, block>>>(offsetH, b_gpu_y, nb, gpu_iv);
    }
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_cpu_y);
    nb = cpu_iv->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel27#" MW_AT_LINE);
      filter_kernel27<<<grid, block>>>(*gpu_x, gpu_iv, nb - 1, gpu_expanded);
    }
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&cpu_iv);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&cpu_rows, 2, &m_emlrtRTEI, true);
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = static_cast<int32_T>(window_idx_0);
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_rows, k, &n_emlrtRTEI);
    cpu_rows->data[0] = 0;
    na = 0;
    profileLoopStart("filter_loop_1", __LINE__, (KH - 2) + 1, "");
    for (k = 0; k <= KH - 2; k++) {
      na++;
      cpu_rows->data[k + 1] = na;
    }
    profileLoopEnd();
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&cpu_convOut, 1, &l_emlrtRTEI, true);
    k = cpu_convOut->size[0];
    cpu_convOut->size[0] = static_cast<int32_T>(OH);
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_convOut, k, &m_emlrtRTEI);
    if (blockDims_idx_0 < 4.294967296E+9) {
      if (blockDims_idx_0 >= 0.0) {
        window_idx_0 = static_cast<uint32_T>(blockDims_idx_0);
      } else {
        window_idx_0 = 0U;
      }
    } else if (blockDims_idx_0 >= 4.294967296E+9) {
      window_idx_0 = MAX_uint32_T;
    } else {
      window_idx_0 = 0U;
    }
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwApplyLaunchParameters(
        computeNumIters(static_cast<int32_T>(OH) - 1),
        dim3(window_idx_0, 1U, 1U),
        dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid, &block);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel28#" MW_AT_LINE);
      filter_kernel28<<<grid, block>>>(gpu_expanded, gpu_rows, *gpu_b,
                                       static_cast<int32_T>(OH) - 1,
                                       gpu_convOut, cpu_b->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    convOut_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&cpu_rows);
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&cpu_expanded);
    if (cpu_zi->size[0] != 0) {
      if (cpu_zi->size[0] == 1) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        cpu_convOut->data[0] += cpu_zi->data[0];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      } else {
        na = cpu_zi->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel32#" MW_AT_LINE);
          filter_kernel32<<<grid, block>>>(*gpu_zi, na - 1, gpu_convOut);
        }
      }
    }
    na = cpu_x->size[0] + cpu_a->size[0];
    profileLoopStart("filter_loop_2", __LINE__, (na - 2) + 1, "");
    for (k = 0; k <= na - 2; k++) {
      nb = static_cast<int32_T>(std::fmin(static_cast<real_T>(k) + 1.0,
                                          static_cast<real_T>(cpu_a->size[0])));
      profileLoopStart("filter_loop_3", __LINE__, (nb - 2) + 1, "");
      for (offsetH = 0; offsetH <= nb - 2; offsetH++) {
        if (convOut_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        }
        if (*a_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
        }
        *a_outdatedOnCpu = false;
        cpu_convOut->data[k] -=
            cpu_convOut->data[(k - offsetH) - 1] * cpu_a->data[offsetH + 1];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        a_needsGpuEnsureCapacity = true;
        b_needsGpuEnsureCapacity = true;
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    if (static_cast<uint32_T>(cpu_x->size[0]) + 1U >
        static_cast<uint32_T>(cpu_convOut->size[0])) {
      na = 0;
      nb = 0;
    } else {
      na = cpu_x->size[0];
      nb = cpu_convOut->size[0];
    }
    k = cpu_zf->size[0];
    cpu_zf->size[0] = nb - na;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_zf, k, &l_emlrtRTEI);
    nb = (nb - na) - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut,
                                    !convOut_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_zf, gpu_zf, true);
    if (convOut_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel29#" MW_AT_LINE);
      filter_kernel29<<<grid, block>>>(gpu_convOut, na, nb, *gpu_zf);
    }
    na = cpu_a->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 3), &grid, &block,
                            2147483647U);
    if (a_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, true);
    }
    *x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel30#" MW_AT_LINE);
      filter_kernel30<<<grid, block>>>(gpu_convOut, *gpu_a, na - 3, *gpu_zf,
                                       cpu_a->size[0U], cpu_x->size[0U]);
    }
    *zf_outdatedOnCpu = true;
    k = cpu_zf->size[0];
    cpu_zf->size[0] = zfSize_idx_0;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_zf, k, &l_emlrtRTEI);
    k = cpu_y->size[0];
    cpu_y->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_y, k, &l_emlrtRTEI);
    nb = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_y, gpu_y, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel31#" MW_AT_LINE);
      filter_kernel31<<<grid, block>>>(gpu_convOut, nb, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&cpu_convOut);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_convOut);
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
  *zf_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace coder

// End of code generation (filter.cu)

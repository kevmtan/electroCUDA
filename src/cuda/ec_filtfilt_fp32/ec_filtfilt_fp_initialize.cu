//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_filtfilt_fp_initialize.cu
//
// Code generation for function 'ec_filtfilt_fp_initialize'
//

// Include files
#include "ec_filtfilt_fp_initialize.h"
#include "_coder_ec_filtfilt_fp_mex.h"
#include "ec_filtfilt_fp_data.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void ec_filtfilt_fp_once();

// Function Definitions
static void ec_filtfilt_fp_once()
{
  mex_InitInfAndNan();
  nvtxRangePushA("#initFcn#ec_filtfilt_fp_once#" MW_AT_LOCATION);
  // Initialize GPU by calling hipFree(nullptr)
  nvtxMarkA("#hipFree#" MW_AT_LINE);
  hipFree(nullptr);
  nvtxRangePop();
}

void ec_filtfilt_fp_initialize()
{
  nvtxRangePushA("#fcn#ec_filtfilt_fp_initialize#" MW_AT_LOCATION);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    nvtxMarkA("#ec_filtfilt_fp_once#" MW_AT_LINE);
    ec_filtfilt_fp_once();
  }
  nvtxRangePop();
}

// End of code generation (ec_filtfilt_fp_initialize.cu)

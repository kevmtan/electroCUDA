#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// eig.cu
//
// Code generation for function 'eig'
//

// Include files
#include "eig.h"
#include "ec_detr_fp_data.h"
#include "ec_detr_fp_emxutil.h"
#include "ec_detr_fp_mexutil.h"
#include "ec_detr_fp_types.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "lapacke.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstddef>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo e_emlrtRTEI{
    56,                                                            // lineNo
    24,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    60,                                                            // lineNo
    28,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    85,                                                            // lineNo
    9,                                                             // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    1,         // lineNo
    30,        // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    89,                                                            // lineNo
    13,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo
    j_emlrtRTEI{
        40,      // lineNo
        37,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo k_emlrtRTEI{
    47,        // lineNo
    20,        // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo
    l_emlrtRTEI{
        1,        // lineNo
        27,       // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo
    m_emlrtRTEI{
        99,      // lineNo
        24,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo n_emlrtRTEI{
    42,                                                              // lineNo
    9,                                                               // colNo
    "schur",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/schur.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    62,        // lineNo
    9,         // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo
    p_emlrtRTEI{
        102,     // lineNo
        21,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    q_emlrtRTEI{
        76,       // lineNo
        22,       // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo r_emlrtRTEI{
    28,                     // lineNo
    9,                      // colNo
    "eigHermitianStandard", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/private/"
    "eigHermitianStandard.m" // pName
};

static emlrtRTEInfo
    s_emlrtRTEI{
        131,     // lineNo
        29,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo t_emlrtRTEI{
    69,                                                              // lineNo
    13,                                                              // colNo
    "schur",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/schur.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    63,        // lineNo
    9,         // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo
    v_emlrtRTEI{
        132,     // lineNo
        29,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo w_emlrtRTEI{
    46,                                                              // lineNo
    9,                                                               // colNo
    "schur",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/schur.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    33,                     // lineNo
    5,                      // colNo
    "eigHermitianStandard", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/private/"
    "eigHermitianStandard.m" // pName
};

static emlrtRTEInfo
    y_emlrtRTEI{
        86,       // lineNo
        9,        // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo
    ab_emlrtRTEI{
        134,     // lineNo
        35,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    bb_emlrtRTEI{
        87,       // lineNo
        9,        // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo cb_emlrtRTEI{
    23,                             // lineNo
    9,                              // colNo
    "eigRealSkewSymmetricStandard", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/private/"
    "eigRealSkewSymmetricStandard.m" // pName
};

static emlrtRTEInfo
    db_emlrtRTEI{
        111,      // lineNo
        29,       // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    eb_emlrtRTEI{
        112,      // lineNo
        29,       // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    fb_emlrtRTEI{
        160,     // lineNo
        13,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo gb_emlrtRTEI{
    24,                             // lineNo
    9,                              // colNo
    "eigRealSkewSymmetricStandard", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/private/"
    "eigRealSkewSymmetricStandard.m" // pName
};

static emlrtRTEInfo
    hb_emlrtRTEI{
        168,     // lineNo
        16,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    ib_emlrtRTEI{
        129,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    jb_emlrtRTEI{
        161,     // lineNo
        13,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo kb_emlrtRTEI{
    27,            // lineNo
    13,            // colNo
    "eigStandard", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/private/"
    "eigStandard.m" // pName
};

static emlrtRTEInfo
    lb_emlrtRTEI{
        130,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo mb_emlrtRTEI{
    1,                                                             // lineNo
    20,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo
    nb_emlrtRTEI{
        99,      // lineNo
        1,       // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    ob_emlrtRTEI{
        131,     // lineNo
        5,       // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    pb_emlrtRTEI{
        132,     // lineNo
        5,       // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    qb_emlrtRTEI{
        134,     // lineNo
        10,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    rb_emlrtRTEI{
        111,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    sb_emlrtRTEI{
        112,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

// Function Definitions
//
//
namespace coder {
void eig(const emxArray_real32_T *A, emxArray_creal32_T *V,
         emxArray_creal32_T *D)
{
  ptrdiff_t ihi_t;
  ptrdiff_t ilo_t;
  emxArray_creal32_T *b_W;
  emxArray_real32_T *U;
  emxArray_real32_T *W;
  emxArray_real32_T *b_A;
  emxArray_real32_T *c_A;
  emxArray_real32_T *d_A;
  emxArray_real32_T *scale;
  emxArray_real32_T *tau;
  emxArray_real32_T *vright;
  emxArray_real32_T *wi;
  emxArray_real32_T *wimag;
  emxArray_real32_T *wr;
  emxArray_real32_T *wreal;
  int32_T j;
  int32_T n;
  real32_T abnrm;
  real32_T rconde;
  real32_T rcondv;
  real32_T vleft;
  nvtxRangePushA("#fcn#eig#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  n = A->size[0];
  j = V->size[0] * V->size[1];
  V->size[0] = A->size[0];
  V->size[1] = A->size[0];
  nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
  emxEnsureCapacity_creal32_T(V, j, &e_emlrtRTEI);
  j = D->size[0] * D->size[1];
  D->size[0] = A->size[0];
  D->size[1] = A->size[0];
  nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
  emxEnsureCapacity_creal32_T(D, j, &f_emlrtRTEI);
  if ((A->size[0] != 0) && (A->size[1] != 0)) {
    int32_T istart;
    int32_T nx;
    boolean_T p;
    nx = A->size[0] * A->size[1];
    p = true;
    profileLoopStart("eig_loop_0", __LINE__, (nx - 1) + 1, "");
    for (istart = 0; istart < nx; istart++) {
      if ((!p) ||
          (std::isinf(A->data[istart]) || std::isnan(A->data[istart]))) {
        p = false;
      }
    }
    profileLoopEnd();
    if (!p) {
      nx = A->size[0];
      istart = A->size[0];
      j = V->size[0] * V->size[1];
      V->size[0] = A->size[0];
      V->size[1] = A->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(V, j, &g_emlrtRTEI);
      profileLoopStart("eig_loop_1", __LINE__, (nx * istart - 1) + 1, "");
      for (j = 0; j < nx * istart; j++) {
        V->data[j].re = rtNaNF;
        V->data[j].im = 0.0F;
      }
      profileLoopEnd();
      nx = A->size[0];
      istart = A->size[0];
      j = D->size[0] * D->size[1];
      D->size[0] = A->size[0];
      D->size[1] = A->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(D, j, &i_emlrtRTEI);
      if (nx * istart - 1 >= 0) {
        std::memset(&D->data[0], 0,
                    static_cast<uint32_T>(nx * istart) * sizeof(creal32_T));
      }
      profileLoopStart("eig_loop_5", __LINE__, (n - 1) + 1, "");
      for (istart = 0; istart < n; istart++) {
        D->data[istart + D->size[0] * istart].re = rtNaNF;
        D->data[istart + D->size[0] * istart].im = 0.0F;
      }
      profileLoopEnd();
    } else {
      int32_T exitg1;
      boolean_T exitg2;
      p = (A->size[0] == A->size[1]);
      if (p) {
        j = 0;
        exitg2 = false;
        nvtxRangePushA("#loop#eig_whileloop_4##" MW_AT_LINE);
        while ((!exitg2) && (j <= A->size[1] - 1)) {
          istart = 0;
          nvtxRangePushA("#loop#eig_whileloop_5##" MW_AT_LINE);
          do {
            exitg1 = 0;
            if (istart <= j) {
              if (!(A->data[istart + A->size[0] * j] ==
                    A->data[j + A->size[0] * istart])) {
                p = false;
                exitg1 = 1;
              } else {
                istart++;
              }
            } else {
              j++;
              exitg1 = 2;
            }
          } while (exitg1 == 0);
          nvtxRangePop();
          if (exitg1 == 1) {
            exitg2 = true;
          }
        }
        nvtxRangePop();
      }
      if (p) {
        ptrdiff_t info_t;
        n = A->size[0];
        nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
        emxInit_real32_T(&b_A, 2, &mb_emlrtRTEI, true);
        j = b_A->size[0] * b_A->size[1];
        b_A->size[0] = A->size[0];
        b_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_A, j, &h_emlrtRTEI);
        profileLoopStart("eig_loop_3", __LINE__,
                         (A->size[0] * A->size[1] - 1) + 1, "");
        for (j = 0; j < A->size[0] * A->size[1]; j++) {
          b_A->data[j] = A->data[j];
        }
        ptrdiff_t n_t;
        profileLoopEnd();
        n_t = (ptrdiff_t)b_A->size[0];
        nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
        emxInit_real32_T(&W, 1, &mb_emlrtRTEI, true);
        j = W->size[0];
        W->size[0] = b_A->size[0];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(W, j, &k_emlrtRTEI);
        info_t =
            LAPACKE_ssyev(102, 'V', 'L', n_t, &b_A->data[0], n_t, &W->data[0]);
        if ((int32_T)info_t < 0) {
          nx = W->size[0];
          j = W->size[0];
          W->size[0] = nx;
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(W, j, &o_emlrtRTEI);
          profileLoopStart("eig_loop_8", __LINE__, (nx - 1) + 1, "");
          for (j = 0; j < nx; j++) {
            W->data[j] = rtNaNF;
          }
          profileLoopEnd();
          nx = b_A->size[0];
          istart = b_A->size[1];
          j = b_A->size[0] * b_A->size[1];
          b_A->size[0] = nx;
          b_A->size[1] = istart;
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_A, j, &u_emlrtRTEI);
          profileLoopStart("eig_loop_15", __LINE__, (nx * istart - 1) + 1, "");
          for (j = 0; j < nx * istart; j++) {
            b_A->data[j] = rtNaNF;
          }
          profileLoopEnd();
        }
        j = D->size[0] * D->size[1];
        D->size[0] = A->size[0];
        D->size[1] = A->size[0];
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(D, j, &r_emlrtRTEI);
        profileLoopStart("eig_loop_9", __LINE__,
                         (A->size[0] * A->size[0] - 1) + 1, "");
        for (j = 0; j < A->size[0] * A->size[0]; j++) {
          D->data[j].re = 0.0F;
          D->data[j].im = 0.0F;
        }
        profileLoopEnd();
        profileLoopStart("eig_loop_11", __LINE__, (n - 1) + 1, "");
        for (istart = 0; istart < n; istart++) {
          D->data[istart + D->size[0] * istart].re = W->data[istart];
          D->data[istart + D->size[0] * istart].im = 0.0F;
        }
        profileLoopEnd();
        nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
        emxFree_real32_T(&W);
        j = V->size[0] * V->size[1];
        V->size[0] = b_A->size[0];
        V->size[1] = b_A->size[1];
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(V, j, &x_emlrtRTEI);
        profileLoopStart("eig_loop_17", __LINE__,
                         (b_A->size[0] * b_A->size[1] - 1) + 1, "");
        for (j = 0; j < b_A->size[0] * b_A->size[1]; j++) {
          V->data[j].re = b_A->data[j];
          V->data[j].im = 0.0F;
        }
        profileLoopEnd();
        nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
        emxFree_real32_T(&b_A);
      } else {
        p = (A->size[0] == A->size[1]);
        if (p) {
          j = 0;
          exitg2 = false;
          nvtxRangePushA("#loop#eig_whileloop_2##" MW_AT_LINE);
          while ((!exitg2) && (j <= A->size[1] - 1)) {
            istart = 0;
            nvtxRangePushA("#loop#eig_whileloop_3##" MW_AT_LINE);
            do {
              exitg1 = 0;
              if (istart <= j) {
                if (!(A->data[istart + A->size[0] * j] ==
                      -A->data[j + A->size[0] * istart])) {
                  p = false;
                  exitg1 = 1;
                } else {
                  istart++;
                }
              } else {
                j++;
                exitg1 = 2;
              }
            } while (exitg1 == 0);
            nvtxRangePop();
            if (exitg1 == 1) {
              exitg2 = true;
            }
          }
          nvtxRangePop();
        }
        if (p) {
          nx = A->size[0] * A->size[1];
          profileLoopStart("eig_loop_2", __LINE__, (nx - 1) + 1, "");
          for (istart = 0; istart < nx; istart++) {
            if ((!p) ||
                (std::isinf(A->data[istart]) || std::isnan(A->data[istart]))) {
              p = false;
            }
          }
          profileLoopEnd();
          nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
          emxInit_real32_T(&d_A, 2, &mb_emlrtRTEI, true);
          nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
          emxInit_real32_T(&U, 2, &mb_emlrtRTEI, true);
          if (!p) {
            uint32_T dv_idx_0;
            uint32_T dv_idx_1;
            dv_idx_0 = static_cast<uint32_T>(A->size[0]);
            dv_idx_1 = static_cast<uint32_T>(A->size[1]);
            j = U->size[0] * U->size[1];
            U->size[0] = A->size[0];
            U->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
            emxEnsureCapacity_real32_T(U, j, &n_emlrtRTEI);
            profileLoopStart("eig_loop_7", __LINE__,
                             (static_cast<int32_T>(dv_idx_0) *
                                  static_cast<int32_T>(dv_idx_1) -
                              1) +
                                 1,
                             "");
            for (j = 0; j < static_cast<int32_T>(dv_idx_0) *
                                static_cast<int32_T>(dv_idx_1);
                 j++) {
              U->data[j] = rtNaNF;
            }
            profileLoopEnd();
            nx = A->size[0];
            if (A->size[0] > 1) {
              istart = 2;
              if (A->size[0] - 2 < A->size[1] - 1) {
                n = A->size[0] - 1;
              } else {
                n = A->size[1];
              }
              profileLoopStart("eig_loop_10", __LINE__, (n - 1) + 1, "");
              for (j = 0; j < n; j++) {
                int64_T b;
                nvtxMarkA("#computeEndIdx#" MW_AT_LINE);
                b = computeEndIdx(static_cast<int64_T>(istart),
                                  static_cast<int64_T>(nx), 1L);
                profileLoopStart("eig_loop_14", __LINE__, b + 1L, "");
                for (int64_T i{0L}; i <= b; i++) {
                  U->data[(static_cast<int32_T>(istart + i) + U->size[0] * j) -
                          1] = 0.0F;
                }
                profileLoopEnd();
                istart++;
              }
              profileLoopEnd();
            }
            dv_idx_0 = static_cast<uint32_T>(A->size[0]);
            dv_idx_1 = static_cast<uint32_T>(A->size[1]);
            j = d_A->size[0] * d_A->size[1];
            d_A->size[0] = A->size[0];
            d_A->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
            emxEnsureCapacity_real32_T(d_A, j, &w_emlrtRTEI);
            profileLoopStart("eig_loop_13", __LINE__,
                             (static_cast<int32_T>(dv_idx_0) *
                                  static_cast<int32_T>(dv_idx_1) -
                              1) +
                                 1,
                             "");
            for (j = 0; j < static_cast<int32_T>(dv_idx_0) *
                                static_cast<int32_T>(dv_idx_1);
                 j++) {
              d_A->data[j] = rtNaNF;
            }
            profileLoopEnd();
          } else {
            ptrdiff_t b_n_t;
            ptrdiff_t e_info_t;
            j = d_A->size[0] * d_A->size[1];
            d_A->size[0] = A->size[0];
            d_A->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
            emxEnsureCapacity_real32_T(d_A, j, &l_emlrtRTEI);
            profileLoopStart("eig_loop_6", __LINE__,
                             (A->size[0] * A->size[1] - 1) + 1, "");
            for (j = 0; j < A->size[0] * A->size[1]; j++) {
              d_A->data[j] = A->data[j];
            }
            profileLoopEnd();
            nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
            emxInit_real32_T(&tau, 1, &mb_emlrtRTEI, true);
            j = tau->size[0];
            tau->size[0] = d_A->size[0] - 1;
            nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
            emxEnsureCapacity_real32_T(tau, j, &q_emlrtRTEI);
            if (d_A->size[0] > 1) {
              ptrdiff_t b_info_t;
              b_info_t =
                  LAPACKE_sgehrd(102, (ptrdiff_t)d_A->size[0], (ptrdiff_t)1,
                                 (ptrdiff_t)d_A->size[0], &d_A->data[0],
                                 (ptrdiff_t)d_A->size[0], &tau->data[0]);
              if ((int32_T)b_info_t != 0) {
                nx = d_A->size[0];
                istart = d_A->size[1];
                j = d_A->size[0] * d_A->size[1];
                d_A->size[0] = nx;
                d_A->size[1] = istart;
                nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
                emxEnsureCapacity_real32_T(d_A, j, &y_emlrtRTEI);
                profileLoopStart("eig_loop_16", __LINE__, (nx * istart - 1) + 1,
                                 "");
                for (j = 0; j < nx * istart; j++) {
                  d_A->data[j] = rtNaNF;
                }
                profileLoopEnd();
                nx = tau->size[0];
                j = tau->size[0];
                tau->size[0] = nx;
                nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
                emxEnsureCapacity_real32_T(tau, j, &bb_emlrtRTEI);
                profileLoopStart("eig_loop_19", __LINE__, (nx - 1) + 1, "");
                for (j = 0; j < nx; j++) {
                  tau->data[j] = rtNaNF;
                }
                profileLoopEnd();
              }
            }
            j = U->size[0] * U->size[1];
            U->size[0] = d_A->size[0];
            U->size[1] = d_A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
            emxEnsureCapacity_real32_T(U, j, &t_emlrtRTEI);
            profileLoopStart("eig_loop_12", __LINE__,
                             (d_A->size[0] * d_A->size[1] - 1) + 1, "");
            for (j = 0; j < d_A->size[0] * d_A->size[1]; j++) {
              U->data[j] = d_A->data[j];
            }
            profileLoopEnd();
            if (A->size[0] == 1) {
              U->data[0] = 1.0F;
            } else {
              ptrdiff_t c_info_t;
              c_info_t =
                  LAPACKE_sorghr(102, (ptrdiff_t)A->size[0], (ptrdiff_t)1,
                                 (ptrdiff_t)A->size[0], &U->data[0],
                                 (ptrdiff_t)A->size[0], &tau->data[0]);
              if ((int32_T)c_info_t != 0) {
                nx = U->size[0];
                istart = U->size[1];
                j = U->size[0] * U->size[1];
                U->size[0] = nx;
                U->size[1] = istart;
                nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
                emxEnsureCapacity_real32_T(U, j, &t_emlrtRTEI);
                profileLoopStart("eig_loop_20", __LINE__, (nx * istart - 1) + 1,
                                 "");
                for (j = 0; j < nx * istart; j++) {
                  U->data[j] = rtNaNF;
                }
                profileLoopEnd();
              }
            }
            nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
            emxFree_real32_T(&tau);
            b_n_t = (ptrdiff_t)d_A->size[0];
            nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
            emxInit_real32_T(&wr, 2, &rb_emlrtRTEI, true);
            j = wr->size[0] * wr->size[1];
            wr->size[0] = 1;
            wr->size[1] = d_A->size[0];
            nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
            emxEnsureCapacity_real32_T(wr, j, &db_emlrtRTEI);
            nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
            emxInit_real32_T(&wi, 2, &sb_emlrtRTEI, true);
            j = wi->size[0] * wi->size[1];
            wi->size[0] = 1;
            wi->size[1] = d_A->size[0];
            nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
            emxEnsureCapacity_real32_T(wi, j, &eb_emlrtRTEI);
            if (d_A->size[0] <= 1) {
              n = 1;
            } else {
              n = d_A->size[0];
            }
            e_info_t = LAPACKE_shseqr(102, 'S', 'V', b_n_t, (ptrdiff_t)1,
                                      (ptrdiff_t)d_A->size[0], &d_A->data[0],
                                      b_n_t, &wr->data[0], &wi->data[0],
                                      &U->data[0], (ptrdiff_t)n);
            nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
            emxFree_real32_T(&wi);
            nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
            emxFree_real32_T(&wr);
            if ((int32_T)e_info_t < 0) {
              nx = d_A->size[0];
              istart = d_A->size[1];
              j = d_A->size[0] * d_A->size[1];
              d_A->size[0] = nx;
              d_A->size[1] = istart;
              nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
              emxEnsureCapacity_real32_T(d_A, j, &ib_emlrtRTEI);
              profileLoopStart("eig_loop_29", __LINE__, (nx * istart - 1) + 1,
                               "");
              for (j = 0; j < nx * istart; j++) {
                d_A->data[j] = rtNaNF;
              }
              profileLoopEnd();
              nx = U->size[0];
              istart = U->size[1];
              j = U->size[0] * U->size[1];
              U->size[0] = nx;
              U->size[1] = istart;
              nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
              emxEnsureCapacity_real32_T(U, j, &lb_emlrtRTEI);
              profileLoopStart("eig_loop_32", __LINE__, (nx * istart - 1) + 1,
                               "");
              for (j = 0; j < nx * istart; j++) {
                U->data[j] = rtNaNF;
              }
              profileLoopEnd();
            }
          }
          n = d_A->size[0];
          j = D->size[0] * D->size[1];
          D->size[0] = d_A->size[0];
          D->size[1] = d_A->size[0];
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(D, j, &cb_emlrtRTEI);
          profileLoopStart("eig_loop_18", __LINE__,
                           (d_A->size[0] * d_A->size[0] - 1) + 1, "");
          for (j = 0; j < d_A->size[0] * d_A->size[0]; j++) {
            D->data[j].re = 0.0F;
            D->data[j].im = 0.0F;
          }
          profileLoopEnd();
          istart = 1;
          nvtxRangePushA("#loop#eig_whileloop_1##" MW_AT_LINE);
          while (istart <= n) {
            if ((istart != n) &&
                (d_A->data[istart + d_A->size[0] * (istart - 1)] != 0.0F)) {
              vleft = std::abs(d_A->data[istart + d_A->size[0] * (istart - 1)]);
              D->data[(istart + D->size[0] * (istart - 1)) - 1].re = 0.0F;
              D->data[(istart + D->size[0] * (istart - 1)) - 1].im = vleft;
              D->data[istart + D->size[0] * istart].re = 0.0F;
              D->data[istart + D->size[0] * istart].im = -vleft;
              istart += 2;
            } else {
              istart++;
            }
          }
          nvtxRangePop();
          j = V->size[0] * V->size[1];
          V->size[0] = U->size[0];
          V->size[1] = U->size[1];
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(V, j, &gb_emlrtRTEI);
          profileLoopStart("eig_loop_22", __LINE__,
                           (U->size[0] * U->size[1] - 1) + 1, "");
          for (j = 0; j < U->size[0] * U->size[1]; j++) {
            V->data[j].re = U->data[j];
            V->data[j].im = 0.0F;
          }
          profileLoopEnd();
          nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
          emxFree_real32_T(&U);
          j = 1;
          n = d_A->size[0];
          nvtxRangePushA("#loop#eig_whileloop_0##" MW_AT_LINE);
          while (j <= n) {
            if ((j != n) && (d_A->data[j + d_A->size[0] * (j - 1)] != 0.0F)) {
              if (d_A->data[j + d_A->size[0] * (j - 1)] < 0.0F) {
                nx = 1;
              } else {
                nx = -1;
              }
              profileLoopStart("eig_loop_28", __LINE__, (n - 1) + 1, "");
              for (istart = 0; istart < n; istart++) {
                vleft = V->data[istart + V->size[0] * (j - 1)].re;
                abnrm = static_cast<real32_T>(nx) *
                        V->data[istart + V->size[0] * j].re;
                if (abnrm == 0.0F) {
                  V->data[istart + V->size[0] * (j - 1)].re =
                      vleft / 1.41421354F;
                  V->data[istart + V->size[0] * (j - 1)].im = 0.0F;
                } else if (vleft == 0.0F) {
                  V->data[istart + V->size[0] * (j - 1)].re = 0.0F;
                  V->data[istart + V->size[0] * (j - 1)].im =
                      abnrm / 1.41421354F;
                } else {
                  V->data[istart + V->size[0] * (j - 1)].re =
                      vleft / 1.41421354F;
                  V->data[istart + V->size[0] * (j - 1)].im =
                      abnrm / 1.41421354F;
                }
                V->data[istart + V->size[0] * j].re =
                    V->data[istart + V->size[0] * (j - 1)].re;
                V->data[istart + V->size[0] * j].im =
                    -V->data[istart + V->size[0] * (j - 1)].im;
              }
              profileLoopEnd();
              j += 2;
            } else {
              j++;
            }
          }
          nvtxRangePop();
          nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
          emxFree_real32_T(&d_A);
        } else {
          ptrdiff_t d_info_t;
          n = A->size[0];
          nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
          emxInit_real32_T(&c_A, 2, &j_emlrtRTEI, true);
          j = c_A->size[0] * c_A->size[1];
          c_A->size[0] = A->size[0];
          c_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(c_A, j, &j_emlrtRTEI);
          profileLoopStart("eig_loop_4", __LINE__,
                           (A->size[0] * A->size[1] - 1) + 1, "");
          for (j = 0; j < A->size[0] * A->size[1]; j++) {
            c_A->data[j] = A->data[j];
          }
          profileLoopEnd();
          nx = A->size[1] - 1;
          nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
          emxInit_real32_T(&scale, 1, &nb_emlrtRTEI, true);
          j = scale->size[0];
          scale->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(scale, j, &m_emlrtRTEI);
          nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
          emxInit_creal32_T(&b_W, 1, &mb_emlrtRTEI, true);
          j = b_W->size[0];
          b_W->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(b_W, j, &p_emlrtRTEI);
          nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
          emxInit_real32_T(&wreal, 1, &ob_emlrtRTEI, true);
          j = wreal->size[0];
          wreal->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(wreal, j, &s_emlrtRTEI);
          nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
          emxInit_real32_T(&wimag, 1, &pb_emlrtRTEI, true);
          j = wimag->size[0];
          wimag->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(wimag, j, &v_emlrtRTEI);
          nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
          emxInit_real32_T(&vright, 2, &qb_emlrtRTEI, true);
          j = vright->size[0] * vright->size[1];
          vright->size[0] = A->size[1];
          vright->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(vright, j, &ab_emlrtRTEI);
          d_info_t = LAPACKE_sgeevx(
              102, 'B', 'N', 'V', 'N', (ptrdiff_t)A->size[1], &c_A->data[0],
              (ptrdiff_t)A->size[0], &wreal->data[0], &wimag->data[0], &vleft,
              (ptrdiff_t)1, &vright->data[0], (ptrdiff_t)A->size[1], &ilo_t,
              &ihi_t, &scale->data[0], &abnrm, &rconde, &rcondv);
          nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
          emxFree_real32_T(&scale);
          nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
          emxFree_real32_T(&c_A);
          if ((int32_T)d_info_t < 0) {
            j = b_W->size[0];
            b_W->size[0] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
            emxEnsureCapacity_creal32_T(b_W, j, &fb_emlrtRTEI);
            profileLoopStart("eig_loop_23", __LINE__, (A->size[1] - 1) + 1, "");
            for (j = 0; j < A->size[1]; j++) {
              b_W->data[j].re = rtNaNF;
              b_W->data[j].im = 0.0F;
            }
            profileLoopEnd();
            nx = A->size[1];
            istart = A->size[1];
            j = V->size[0] * V->size[1];
            V->size[0] = A->size[1];
            V->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
            emxEnsureCapacity_creal32_T(V, j, &jb_emlrtRTEI);
            profileLoopStart("eig_loop_26", __LINE__, (nx * istart - 1) + 1,
                             "");
            for (j = 0; j < nx * istart; j++) {
              V->data[j].re = rtNaNF;
              V->data[j].im = 0.0F;
            }
            profileLoopEnd();
          } else {
            profileLoopStart("eig_loop_21", __LINE__, nx + 1, "");
            for (istart = 0; istart <= nx; istart++) {
              b_W->data[istart].re = wreal->data[istart];
              b_W->data[istart].im = wimag->data[istart];
            }
            profileLoopEnd();
            j = V->size[0] * V->size[1];
            V->size[0] = vright->size[0];
            V->size[1] = vright->size[1];
            nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
            emxEnsureCapacity_creal32_T(V, j, &hb_emlrtRTEI);
            profileLoopStart("eig_loop_24", __LINE__,
                             (vright->size[0] * vright->size[1] - 1) + 1, "");
            for (j = 0; j < vright->size[0] * vright->size[1]; j++) {
              V->data[j].re = vright->data[j];
              V->data[j].im = 0.0F;
            }
            profileLoopEnd();
            profileLoopStart("eig_loop_25", __LINE__, (nx - 1) + 1, "");
            for (istart = 0; istart < nx; istart++) {
              if ((wimag->data[istart] > 0.0F) &&
                  (wimag->data[istart + 1] < 0.0F)) {
                profileLoopStart("eig_loop_27", __LINE__, nx + 1, "");
                for (j = 0; j <= nx; j++) {
                  vleft = V->data[j + V->size[0] * istart].re;
                  abnrm = V->data[j + V->size[0] * (istart + 1)].re;
                  V->data[j + V->size[0] * istart].im = abnrm;
                  V->data[j + V->size[0] * (istart + 1)].re = vleft;
                  V->data[j + V->size[0] * (istart + 1)].im = -abnrm;
                }
                profileLoopEnd();
              }
            }
            profileLoopEnd();
          }
          nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
          emxFree_real32_T(&vright);
          nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
          emxFree_real32_T(&wimag);
          nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
          emxFree_real32_T(&wreal);
          j = D->size[0] * D->size[1];
          D->size[0] = A->size[0];
          D->size[1] = A->size[0];
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(D, j, &kb_emlrtRTEI);
          profileLoopStart("eig_loop_30", __LINE__,
                           (A->size[0] * A->size[0] - 1) + 1, "");
          for (j = 0; j < A->size[0] * A->size[0]; j++) {
            D->data[j].re = 0.0F;
            D->data[j].im = 0.0F;
          }
          profileLoopEnd();
          profileLoopStart("eig_loop_31", __LINE__, (n - 1) + 1, "");
          for (istart = 0; istart < n; istart++) {
            D->data[istart + D->size[0] * istart] = b_W->data[istart];
          }
          profileLoopEnd();
          nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
          emxFree_creal32_T(&b_W);
        }
      }
    }
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (eig.cu)

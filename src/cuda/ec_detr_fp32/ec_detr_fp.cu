#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_detr_fp.cu
//
// Code generation for function 'ec_detr_fp'
//

// Include files
#include "ec_detr_fp.h"
#include "ec_detr_fp_data.h"
#include "ec_detr_fp_emxutil.h"
#include "ec_detr_fp_mexutil.h"
#include "ec_detr_fp_types.h"
#include "eig.h"
#include "mrdivide_helper.h"
#include "mtimes.h"
#include "rt_nonfinite.h"
#include "sum.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hip/hip_math_constants.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo ub_emlrtRTEI{
    32,                                                       // lineNo
    24,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo vb_emlrtRTEI{
    58,                                                       // lineNo
    23,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo wb_emlrtRTEI{
    41,                                                       // lineNo
    41,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo xb_emlrtRTEI{
    41,                                                       // lineNo
    49,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo yb_emlrtRTEI{
    80,                                                       // lineNo
    20,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo ac_emlrtRTEI{
    42,                                                       // lineNo
    28,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo bc_emlrtRTEI{
    124,                                                      // lineNo
    10,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo cc_emlrtRTEI{
    109,                                                      // lineNo
    1,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo dc_emlrtRTEI{
    110,                                                      // lineNo
    1,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo
    ec_emlrtRTEI{
        44,       // lineNo
        36,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo fc_emlrtRTEI{
    112,                                                      // lineNo
    1,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo gc_emlrtRTEI{
    109,                                                           // lineNo
    24,                                                            // colNo
    "diag",                                                        // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/elmat/diag.m" // pName
};

static emlrtRTEInfo hc_emlrtRTEI{
    100,                                                           // lineNo
    5,                                                             // colNo
    "diag",                                                        // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/elmat/diag.m" // pName
};

static emlrtRTEInfo ic_emlrtRTEI{
    1,                                                        // lineNo
    24,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo jc_emlrtRTEI{
    114,                                                      // lineNo
    5,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo kc_emlrtRTEI{
    114,                                                      // lineNo
    1,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo lc_emlrtRTEI{
    115,                                                      // lineNo
    6,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo mc_emlrtRTEI{
    108,                                                      // lineNo
    5,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo
    nc_emlrtRTEI{
        94,                  // lineNo
        5,                   // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo
    oc_emlrtRTEI{
        44,       // lineNo
        48,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo pc_emlrtRTEI{
    83,                                                       // lineNo
    5,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo qc_emlrtRTEI{
    86,                                                       // lineNo
    5,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    1,                                                        // lineNo
    15,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo sc_emlrtRTEI{
    37,                                                       // lineNo
    5,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    59,                                                       // lineNo
    1,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    80,                                                       // lineNo
    1,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    83,                                                       // lineNo
    22,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo wc_emlrtRTEI{
    111,                                                      // lineNo
    13,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo xc_emlrtRTEI{
    111,                                                      // lineNo
    2,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo yc_emlrtRTEI{
    113,                                                      // lineNo
    10,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo ad_emlrtRTEI{
    114,                                                      // lineNo
    9,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo bd_emlrtRTEI{
    116,                                                      // lineNo
    6,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo cd_emlrtRTEI{
    116,                                                      // lineNo
    16,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo dd_emlrtRTEI{
    117,                                                      // lineNo
    8,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo ed_emlrtRTEI{
    117,                                                      // lineNo
    5,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    88,                                                       // lineNo
    7,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo
    gd_emlrtRTEI{
        126,       // lineNo
        9,         // colNo
        "vvarstd", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/datafun/private/"
        "vvarstd.m" // pName
    };

static emlrtRTEInfo hd_emlrtRTEI{
    124,                                                      // lineNo
    6,                                                        // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

static emlrtRTEInfo
    id_emlrtRTEI{
        94,                  // lineNo
        9,                   // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo wd_emlrtRTEI{
    41,                                                       // lineNo
    10,                                                       // colNo
    "ec_detr_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_detr_fp.m" // pName
};

// Function Declarations
static void binary_expand_op(emxArray_real32_T *in1,
                             const emxArray_real32_T *in2,
                             const emxArray_real32_T *in3,
                             const emxArray_real32_T *in4);

static void binary_expand_op_1(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2,
                               const emxArray_real32_T *in3, real32_T in4);

static void binary_expand_op_2(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2, real32_T in3,
                               const emxArray_real32_T *in4);

static void binary_expand_op_3(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2,
                               const emxArray_real32_T *in3);

static void binary_expand_op_4(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2,
                               const emxArray_real32_T *in3);

static void binary_expand_op_5(emxArray_real32_T *in1, int32_T in2,
                               const emxArray_real32_T *in3);

static __global__ void ec_detr_fp_kernel1(const int32_T nFrames,
                                          emxArray_real_T lin);

static __global__ void
ec_detr_fp_kernel10(const emxArray_real32_T fv1, const emxArray_real32_T regs,
                    const int32_T b_regs, const int32_T c_regs,
                    emxArray_real32_T d_regs, int32_T regs_dim0,
                    int32_T b_regs_dim0);

static __global__ void
ec_detr_fp_kernel11(const real32_T y, const emxArray_real32_T r,
                    const emxArray_real32_T regs, const int32_T b_regs,
                    const int32_T c_regs, emxArray_real32_T b_r, int32_T r_dim0,
                    int32_T regs_dim0);

static __global__ void ec_detr_fp_kernel12(const emxArray_real32_T fv1,
                                           const emxArray_real32_T r,
                                           const int32_T b_r, const int32_T c_r,
                                           emxArray_real32_T r1,
                                           int32_T r1_dim0, int32_T r_dim0);

static __global__ void ec_detr_fp_kernel13(const emxArray_real32_T r1,
                                           const int32_T b_r1,
                                           const int32_T c_r1,
                                           emxArray_real32_T r2,
                                           int32_T r2_dim0, int32_T r1_dim0);

static __global__ void ec_detr_fp_kernel14(const emxArray_creal32_T V,
                                           const int32_T b_V,
                                           emxArray_real32_T c_V);

static __global__ void ec_detr_fp_kernel15(const emxArray_int32_T iv,
                                           const emxArray_real32_T V,
                                           const int32_T b, const int32_T c,
                                           emxArray_real32_T b_V,
                                           int32_T V_dim0, int32_T b_V_dim0);

static __global__ void ec_detr_fp_kernel16(const emxArray_real32_T V,
                                           const int32_T b_V,
                                           emxArray_real32_T c_V);

static __global__ void ec_detr_fp_kernel17(const emxArray_real32_T fv1,
                                           const real32_T mn,
                                           const emxArray_real32_T fv,
                                           const int32_T b_fv,
                                           emxArray_real32_T A);

static __global__ void ec_detr_fp_kernel18(const int32_T b,
                                           emxArray_real32_T y);

static __global__ void ec_detr_fp_kernel19(const emxArray_real32_T r1,
                                           const int32_T b_r1,
                                           const int32_T c_r1,
                                           emxArray_real32_T r3,
                                           int32_T r3_dim0, int32_T r1_dim0);

static __global__ void ec_detr_fp_kernel2(emxArray_real_T lin);

static __global__ void ec_detr_fp_kernel20(const emxArray_real32_T y,
                                           const int32_T b_y,
                                           emxArray_real32_T c_y);

static __global__ void ec_detr_fp_kernel21(const real32_T mn,
                                           const emxArray_real32_T fv,
                                           const emxArray_real32_T fv2,
                                           const int32_T b_fv2,
                                           emxArray_real32_T z);

static __global__ void ec_detr_fp_kernel22(const emxArray_real32_T fv1,
                                           const emxArray_real32_T z,
                                           const emxArray_real32_T fv,
                                           const int32_T b_fv,
                                           emxArray_real32_T d);

static __global__ void ec_detr_fp_kernel23(const emxArray_real32_T d,
                                           const int32_T b,
                                           emxArray_real32_T y);

static __global__ void ec_detr_fp_kernel24(const real32_T xbar,
                                           const emxArray_real32_T y,
                                           const int32_T b,
                                           emxArray_real32_T fv1);

static __global__ void ec_detr_fp_kernel25(const emxArray_real32_T d,
                                           real32_T *y);

static __global__ void ec_detr_fp_kernel26(const real32_T xbar,
                                           const emxArray_real32_T d,
                                           const int32_T vlen,
                                           emxArray_real32_T absdiff);

static __global__ void ec_detr_fp_kernel27(const emxArray_creal32_T D,
                                           const int32_T b,
                                           emxArray_creal32_T d,
                                           int32_T D_dim0);

static __global__ void ec_detr_fp_kernel28(const emxArray_real32_T z,
                                           const int32_T ch, const int32_T b,
                                           emxArray_real32_T x, int32_T x_dim0);

static __global__ void ec_detr_fp_kernel29(const emxArray_real32_T fv1,
                                           const int32_T ch, const int32_T b,
                                           emxArray_real32_T w, int32_T w_dim0);

static __global__ void ec_detr_fp_kernel3(const real_T d2scaled,
                                          const int32_T b, emxArray_real_T lin,
                                          int32_T lin_dim1);

static __global__ void
ec_detr_fp_kernel30(const int32_T ch, const emxArray_real32_T w,
                    const int32_T b_w, emxArray_boolean_T x, int32_T w_dim0);

static __global__ void ec_detr_fp_kernel4(emxArray_real_T lin,
                                          int32_T lin_dim1);

static __global__ void ec_detr_fp_kernel5(const int32_T k,
                                          const emxArray_real_T lin,
                                          const int32_T vlen, const int32_T b,
                                          emxArray_real32_T regs,
                                          int32_T regs_dim0);

static __global__ void ec_detr_fp_kernel6(const int32_T ch,
                                          const emxArray_real32_T x,
                                          const int32_T b_x,
                                          emxArray_real32_T fv, int32_T x_dim0);

static __global__ void
ec_detr_fp_kernel7(const int32_T ch, const emxArray_real32_T w,
                   const int32_T b_w, emxArray_real32_T fv1, int32_T w_dim0);

static __global__ void ec_detr_fp_kernel8(const emxArray_real32_T fv1,
                                          const emxArray_real32_T fv,
                                          const int32_T b_fv,
                                          emxArray_real32_T x);

static __global__ void ec_detr_fp_kernel9(const emxArray_real32_T fv1,
                                          const emxArray_real32_T fv,
                                          const int32_T b_fv,
                                          emxArray_real32_T x);

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy);

static void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu,
                                           boolean_T needsCopy);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy);

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu);

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu);

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu);

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static void times(emxArray_real32_T *in1, const emxArray_real32_T *in2,
                  const emxArray_real32_T *in3);

// Function Definitions
static void binary_expand_op(emxArray_real32_T *in1,
                             const emxArray_real32_T *in2,
                             const emxArray_real32_T *in3,
                             const emxArray_real32_T *in4)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  int32_T stride_2_0;
  nvtxRangePushA("#fcn#binary_expand_op#" MW_AT_LOCATION);
  i = in1->size[0];
  if (in4->size[0] == 1) {
    if (in3->size[0] == 1) {
      in1->size[0] = in2->size[0];
    } else {
      in1->size[0] = in3->size[0];
    }
  } else {
    in1->size[0] = in4->size[0];
  }
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(in1, i, &qc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  stride_2_0 = (in4->size[0] != 1);
  if (in4->size[0] == 1) {
    if (in3->size[0] == 1) {
      b = in2->size[0];
    } else {
      b = in3->size[0];
    }
  } else {
    b = in4->size[0];
  }
  profileLoopStart("binary_expand_op_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    in1->data[i] = (in2->data[i * stride_0_0] - in3->data[i * stride_1_0]) *
                   in4->data[i * stride_2_0];
  }
  profileLoopEnd();
  nvtxRangePop();
}

static void binary_expand_op_1(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2,
                               const emxArray_real32_T *in3, real32_T in4)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#binary_expand_op_1#" MW_AT_LOCATION);
  i = in1->size[0];
  if (in3->size[0] == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3->size[0];
  }
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(in1, i, &pc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  if (in3->size[0] == 1) {
    b = in2->size[0];
  } else {
    b = in3->size[0];
  }
  profileLoopStart("binary_expand_op_1_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    in1->data[i] =
        in2->data[i * stride_0_0] +
        (in3->data[i * stride_1_0] - (in3->data[i * stride_1_0] - in4));
  }
  profileLoopEnd();
  nvtxRangePop();
}

static void binary_expand_op_2(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2, real32_T in3,
                               const emxArray_real32_T *in4)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  nvtxRangePushA("#fcn#binary_expand_op_2#" MW_AT_LOCATION);
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (in4->size[0] == 1) {
    in1->size[1] = in2->size[0];
  } else {
    in1->size[1] = in4->size[0];
  }
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(in1, i, &mc_emlrtRTEI);
  stride_0_1 = (in2->size[0] != 1);
  stride_1_1 = (in4->size[0] != 1);
  if (in4->size[0] == 1) {
    b = in2->size[0];
  } else {
    b = in4->size[0];
  }
  profileLoopStart("binary_expand_op_2_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    in1->data[i] =
        (in2->data[i * stride_0_1] - in3) * in4->data[i * stride_1_1];
  }
  profileLoopEnd();
  nvtxRangePop();
}

static void binary_expand_op_3(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2,
                               const emxArray_real32_T *in3)
{
  int32_T i;
  int32_T in3_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#binary_expand_op_3#" MW_AT_LOCATION);
  in3_idx_0 = in3->size[0];
  i = in1->size[0] * in1->size[1];
  if (in3_idx_0 == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3_idx_0;
  }
  in1->size[1] = in2->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(in1, i, &dc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3_idx_0 != 1);
  profileLoopStart("binary_expand_op_3_loop_0", __LINE__,
                   (in2->size[1] - 1) + 1, "");
  for (i = 0; i < in2->size[1]; i++) {
    int32_T b;
    if (in3_idx_0 == 1) {
      b = in2->size[0];
    } else {
      b = in3_idx_0;
    }
    profileLoopStart("binary_expand_op_3_loop_1", __LINE__, (b - 1) + 1, "");
    for (int32_T i1{0}; i1 < b; i1++) {
      in1->data[i1 + in1->size[0] * i] =
          in2->data[i1 * stride_0_0 + in2->size[0] * i] *
          in3->data[i1 * stride_1_0];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxRangePop();
}

static void binary_expand_op_4(emxArray_real32_T *in1,
                               const emxArray_real32_T *in2,
                               const emxArray_real32_T *in3)
{
  emxArray_real32_T *b_in2;
  int32_T i;
  int32_T in3_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#binary_expand_op_4#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  in3_idx_0 = in3->size[0];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_in2, 2, &bc_emlrtRTEI, true);
  i = b_in2->size[0] * b_in2->size[1];
  if (in3_idx_0 == 1) {
    b_in2->size[0] = in2->size[0];
  } else {
    b_in2->size[0] = in3_idx_0;
  }
  b_in2->size[1] = in2->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(b_in2, i, &bc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3_idx_0 != 1);
  profileLoopStart("binary_expand_op_4_loop_0", __LINE__,
                   (in2->size[1] - 1) + 1, "");
  for (i = 0; i < in2->size[1]; i++) {
    int32_T b;
    if (in3_idx_0 == 1) {
      b = in2->size[0];
    } else {
      b = in3_idx_0;
    }
    profileLoopStart("binary_expand_op_4_loop_1", __LINE__, (b - 1) + 1, "");
    for (int32_T i1{0}; i1 < b; i1++) {
      b_in2->data[i1 + b_in2->size[0] * i] =
          in2->data[i1 * stride_0_0 + in2->size[0] * i] *
          in3->data[i1 * stride_1_0];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#sum#" MW_AT_LINE);
  coder::sum(b_in2, in1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_in2);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

static void binary_expand_op_5(emxArray_real32_T *in1, int32_T in2,
                               const emxArray_real32_T *in3)
{
  emxArray_real32_T *b_in1;
  int32_T i;
  int32_T in1_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#binary_expand_op_5#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  in1_idx_0 = in1->size[0];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_in1, 1, &wd_emlrtRTEI, true);
  i = b_in1->size[0];
  b_in1->size[0] = in1_idx_0;
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(b_in1, i, &wd_emlrtRTEI);
  stride_0_0 = (in1->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  profileLoopStart("binary_expand_op_5_loop_0", __LINE__, (in1_idx_0 - 1) + 1,
                   "");
  for (i = 0; i < in1_idx_0; i++) {
    b_in1->data[i] = in1->data[i * stride_0_0 + in1->size[0] * in2] -
                     in3->data[i * stride_1_0];
  }
  profileLoopEnd();
  profileLoopStart("binary_expand_op_5_loop_1", __LINE__,
                   (b_in1->size[0] - 1) + 1, "");
  for (i = 0; i < b_in1->size[0]; i++) {
    in1->data[i + in1->size[0] * in2] = b_in1->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_in1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

static __global__
    __launch_bounds__(32, 1) void ec_detr_fp_kernel1(const int32_T nFrames,
                                                     emxArray_real_T lin)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    lin.data[nFrames - 1] = 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel10(
    const emxArray_real32_T fv1, const emxArray_real32_T regs,
    const int32_T b_regs, const int32_T c_regs, emxArray_real32_T d_regs,
    int32_T regs_dim0, int32_T b_regs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_regs) + 1UL) *
                (static_cast<uint64_T>(b_regs) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_regs) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_regs) + 1UL));
    d_regs.data[k + regs_dim0 * i] =
        regs.data[k + b_regs_dim0 * i] * fv1.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel11(
    const real32_T y, const emxArray_real32_T r, const emxArray_real32_T regs,
    const int32_T b_regs, const int32_T c_regs, emxArray_real32_T b_r,
    int32_T r_dim0, int32_T regs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_regs) + 1UL) *
                (static_cast<uint64_T>(b_regs) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_regs) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_regs) + 1UL));
    b_r.data[k + r_dim0 * i] = regs.data[k + regs_dim0 * i] - r.data[i] / y;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel12(
    const emxArray_real32_T fv1, const emxArray_real32_T r, const int32_T b_r,
    const int32_T c_r, emxArray_real32_T r1, int32_T r1_dim0, int32_T r_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_r) + 1UL) * (static_cast<uint64_T>(b_r) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r) + 1UL));
    r1.data[k + r1_dim0 * i] = r.data[k + r_dim0 * i] * fv1.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel13(
    const emxArray_real32_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_real32_T r2, int32_T r2_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r1) + 1UL));
    r2.data[k + r2_dim0 * i] = r1.data[i + r1_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel14(
    const emxArray_creal32_T V, const int32_T b_V, emxArray_real32_T c_V)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_V);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_V.data[i] = V.data[i].re;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel15(
    const emxArray_int32_T iv, const emxArray_real32_T V, const int32_T b,
    const int32_T c, emxArray_real32_T b_V, int32_T V_dim0, int32_T b_V_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b) + 1UL));
    b_V.data[k + V_dim0 * i] = V.data[k + b_V_dim0 * iv.data[i]];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel16(
    const emxArray_real32_T V, const int32_T b_V, emxArray_real32_T c_V)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_V);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_V.data[i] = V.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel17(
    const emxArray_real32_T fv1, const real32_T mn, const emxArray_real32_T fv,
    const int32_T b_fv, emxArray_real32_T A)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_fv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    A.data[i] = (fv.data[i] - mn) * fv1.data[i];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_detr_fp_kernel18(const int32_T b,
                                                        emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    y.data[i] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel19(
    const emxArray_real32_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_real32_T r3, int32_T r3_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r1) + 1UL));
    r3.data[k + r3_dim0 * i] = r1.data[i + r1_dim0 * k];
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_detr_fp_kernel2(emxArray_real_T lin)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    lin.data[0] = -1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel20(
    const emxArray_real32_T y, const int32_T b_y, emxArray_real32_T c_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_y.data[i] = y.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel21(
    const real32_T mn, const emxArray_real32_T fv, const emxArray_real32_T fv2,
    const int32_T b_fv2, emxArray_real32_T z)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_fv2);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    z.data[i] = fv2.data[i] + (fv.data[i] - (fv.data[i] - mn));
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel22(
    const emxArray_real32_T fv1, const emxArray_real32_T z,
    const emxArray_real32_T fv, const int32_T b_fv, emxArray_real32_T d)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_fv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    d.data[i] = (fv.data[i] - z.data[i]) * fv1.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel23(
    const emxArray_real32_T d, const int32_T b, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = fabsf(d.data[k]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel24(
    const real32_T xbar, const emxArray_real32_T y, const int32_T b,
    emxArray_real32_T fv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    if (y.data[k] > xbar) {
      fv1.data[k] = 0.0F;
    }
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_detr_fp_kernel25(const emxArray_real32_T d,
                                                      real32_T *y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    if ((static_cast<boolean_T>(!static_cast<int32_T>(isinf(d.data[0])))) &&
        (static_cast<boolean_T>(!static_cast<int32_T>(isnan(d.data[0]))))) {
      *y = 0.0F;
    } else {
      *y = HIP_NAN_F;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel26(
    const real32_T xbar, const emxArray_real32_T d, const int32_T vlen,
    emxArray_real32_T absdiff)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(vlen);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    absdiff.data[k] = fabsf(d.data[k] - xbar);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel27(
    const emxArray_creal32_T D, const int32_T b, emxArray_creal32_T d,
    int32_T D_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    d.data[k] = D.data[k + D_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel28(
    const emxArray_real32_T z, const int32_T ch, const int32_T b,
    emxArray_real32_T x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i + x_dim0 * ch] -= z.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel29(
    const emxArray_real32_T fv1, const int32_T ch, const int32_T b,
    emxArray_real32_T w, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    w.data[i + w_dim0 * ch] = fv1.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel3(
    const real_T d2scaled, const int32_T b, emxArray_real_T lin,
    int32_T lin_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    lin.data[k + 1] =
        static_cast<real_T>((((k + 2) << 1) - lin_dim1) - 1) * d2scaled;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel30(
    const int32_T ch, const emxArray_real32_T w, const int32_T b_w,
    emxArray_boolean_T x, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_w);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = static_cast<boolean_T>(
        !static_cast<int32_T>(w.data[i + w_dim0 * ch] != 0.0F));
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_detr_fp_kernel4(emxArray_real_T lin,
                                                     int32_T lin_dim1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    lin.data[lin_dim1 >> 1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel5(
    const int32_T k, const emxArray_real_T lin, const int32_T vlen,
    const int32_T b, emxArray_real32_T regs, int32_T regs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    b_k = static_cast<int32_T>(idx);
    regs.data[b_k + regs_dim0 * (vlen - 1)] = powf(
        static_cast<real32_T>(lin.data[b_k]), static_cast<real32_T>(k) + 1.0F);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel6(
    const int32_T ch, const emxArray_real32_T x, const int32_T b_x,
    emxArray_real32_T fv, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    fv.data[i] = x.data[i + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel7(
    const int32_T ch, const emxArray_real32_T w, const int32_T b_w,
    emxArray_real32_T fv1, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_w);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    fv1.data[i] = w.data[i + w_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel8(
    const emxArray_real32_T fv1, const emxArray_real32_T fv, const int32_T b_fv,
    emxArray_real32_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_fv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = fv.data[i] * fv1.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_fp_kernel9(
    const emxArray_real32_T fv1, const emxArray_real32_T fv, const int32_T b_fv,
    emxArray_real32_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_fv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = fv.data[i] * fv1.data[i];
  }
}

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  boolean_T *newData;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_boolean_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxEnsureCapacity_boolean_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(boolean_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(boolean_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

static void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu,
                                           boolean_T needsCopy)
{
  creal32_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxEnsureCapacity_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(creal32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(creal32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_boolean_T#" MW_AT_LOCATION);
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

static void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_creal32_T#" MW_AT_LOCATION);
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real_T#" MW_AT_LOCATION);
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyCpuToGpu_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (cpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(gpu->data, cpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                   hipMemcpyHostToDevice),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_boolean_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyGpuToCpu_boolean_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(cpu->data, gpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(boolean_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyGpuToCpu_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(cpu->data, gpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_boolean_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_boolean_T));
  nvtxRangePop();
}

static void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_creal32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_creal32_T));
  nvtxRangePop();
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real_T));
  nvtxRangePop();
}

static void times(emxArray_real32_T *in1, const emxArray_real32_T *in2,
                  const emxArray_real32_T *in3)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#times#" MW_AT_LOCATION);
  i = in1->size[0];
  if (in3->size[0] == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3->size[0];
  }
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(in1, i, &bc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  if (in3->size[0] == 1) {
    b = in2->size[0];
  } else {
    b = in3->size[0];
  }
  profileLoopStart("times_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    in1->data[i] = in2->data[i * stride_0_0] * in3->data[i * stride_1_0];
  }
  profileLoopEnd();
  nvtxRangePop();
}

//
// function [x,w,olPct] = ec_detr_fp(x,w,ord,thr,itr)
void ec_detr_fp(emxArray_real32_T *cpu_x, emxArray_real32_T *cpu_w,
                const real32_T ord_data[], const int32_T ord_size[2],
                const real32_T thr_data[], const int32_T[2],
                const real32_T itr_data[], const int32_T[2],
                real_T olPct_data[], int32_T olPct_size[2])
{
  dim3 block;
  dim3 grid;
  emxArray_boolean_T d_gpu_x;
  emxArray_boolean_T *b_cpu_x;
  emxArray_creal32_T b_gpu_d;
  emxArray_creal32_T gpu_D;
  emxArray_creal32_T gpu_V;
  emxArray_creal32_T *cpu_D;
  emxArray_creal32_T *cpu_V;
  emxArray_creal32_T *cpu_d;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T *cpu_iv;
  emxArray_real32_T b_gpu_V;
  emxArray_real32_T b_gpu_r;
  emxArray_real32_T b_gpu_regs;
  emxArray_real32_T b_gpu_x;
  emxArray_real32_T b_gpu_y;
  emxArray_real32_T c_gpu_V;
  emxArray_real32_T c_gpu_x;
  emxArray_real32_T c_gpu_y;
  emxArray_real32_T d_gpu_y;
  emxArray_real32_T e_gpu_y;
  emxArray_real32_T f_gpu_y;
  emxArray_real32_T gpu_A;
  emxArray_real32_T gpu_absdiff;
  emxArray_real32_T gpu_d;
  emxArray_real32_T gpu_fv;
  emxArray_real32_T gpu_fv1;
  emxArray_real32_T gpu_fv2;
  emxArray_real32_T gpu_r;
  emxArray_real32_T gpu_r1;
  emxArray_real32_T gpu_r2;
  emxArray_real32_T gpu_r3;
  emxArray_real32_T gpu_r4;
  emxArray_real32_T gpu_regs;
  emxArray_real32_T gpu_w;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_y;
  emxArray_real32_T gpu_z;
  emxArray_real32_T *b_cpu_V;
  emxArray_real32_T *b_cpu_d;
  emxArray_real32_T *b_cpu_r;
  emxArray_real32_T *b_cpu_regs;
  emxArray_real32_T *b_cpu_y;
  emxArray_real32_T *c_cpu_V;
  emxArray_real32_T *c_cpu_x;
  emxArray_real32_T *c_cpu_y;
  emxArray_real32_T *cpu_A;
  emxArray_real32_T *cpu_absdiff;
  emxArray_real32_T *cpu_fv;
  emxArray_real32_T *cpu_fv1;
  emxArray_real32_T *cpu_fv2;
  emxArray_real32_T *cpu_r;
  emxArray_real32_T *cpu_r1;
  emxArray_real32_T *cpu_r2;
  emxArray_real32_T *cpu_r3;
  emxArray_real32_T *cpu_r4;
  emxArray_real32_T *cpu_regs;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *cpu_z;
  emxArray_real32_T *d_cpu_x;
  emxArray_real32_T *d_cpu_y;
  emxArray_real32_T *e_cpu_y;
  emxArray_real32_T *f_cpu_y;
  emxArray_real_T gpu_lin;
  emxArray_real_T *cpu_lin;
  emxArray_real_T *olPct;
  int32_T b_i;
  int32_T i;
  int32_T i5;
  int32_T nFrames;
  real32_T g_cpu_y;
  real32_T x;
  real32_T *g_gpu_y;
  boolean_T A_outdatedOnCpu;
  boolean_T b_x_outdatedOnCpu;
  boolean_T c_x_outdatedOnCpu;
  boolean_T d_outdatedOnCpu;
  boolean_T fv2_outdatedOnCpu;
  boolean_T fv2_outdatedOnGpu;
  boolean_T iv_outdatedOnGpu;
  boolean_T lin_needsGpuEnsureCapacity;
  boolean_T r1_outdatedOnCpu;
  boolean_T r1_outdatedOnGpu;
  boolean_T r_outdatedOnGpu;
  boolean_T w_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnCpu;
  boolean_T y_outdatedOnCpu;
  boolean_T y_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_detr_fp#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&g_gpu_y, 4UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_r4);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_r3);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_V);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_r2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_regs);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_r1);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_absdiff);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&f_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_d);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_fv2);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&e_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_A);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&b_gpu_d);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_V);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_D);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_V);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_r);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxReset_boolean_T#" MW_AT_LINE);
  gpuEmxReset_boolean_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_z);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_fv1);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_fv);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_lin);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_regs);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_w);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_x);
  r1_outdatedOnCpu = false;
  d_outdatedOnCpu = false;
  A_outdatedOnCpu = false;
  iv_outdatedOnGpu = false;
  y_outdatedOnGpu = false;
  c_x_outdatedOnCpu = false;
  b_x_outdatedOnCpu = false;
  fv2_outdatedOnCpu = false;
  w_outdatedOnGpu = true;
  x_outdatedOnCpu = false;
  fv2_outdatedOnGpu = true;
  y_outdatedOnCpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  Robust detrending - wrapper for GPU processing (CUDA source code)
  //
  //  INPUTS:
  //     see 'input validation' section below
  //  OUTPUTS:
  //     y = detrended data
  //     w = updated weights
  //     r = basis matrix used
  //
  //  This code was adapted from NoiseTools (audition.ens.fr/adc/NoiseTools)
  //
  //  Modified by: Kevin Tan, 2022 (kevmtan.github.io)
  //     - Added GPU support
  //     - Replaced bsxfun with implicit expansion operators (much faster)
  //     - Miscellaneous compute & readability enhancements
  //  Input validation
  // 'ec_detr_fp:20' x (:,:){mustBeFloat}
  // 'ec_detr_fp:21' w (:,:){mustBeFloat}
  // 'ec_detr_fp:22' ord (1,:){mustBeFloat}
  // 'ec_detr_fp:23' thr (1,:){mustBeFloat}
  // 'ec_detr_fp:24' itr (1,:){mustBeFloat}
  //  Main
  // 'ec_detr_fp:28' coder.gpu.kernelfun;
  //  trigger CUDA kernel generation
  // 'ec_detr_fp:29' nFrames = size(x,1);
  nFrames = cpu_x->size[0];
  // 'ec_detr_fp:30' nChs = size(x,2);
  // 'ec_detr_fp:31' reps = numel(ord);
  // 'ec_detr_fp:32' olPct = coder.nullcopy(nan(nChs,reps));
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&olPct, 2, &rc_emlrtRTEI, true);
  i = olPct->size[0] * olPct->size[1];
  olPct->size[0] = cpu_x->size[1];
  olPct->size[1] = ord_size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(olPct, i, &ub_emlrtRTEI);
  //  Detrend by polynomial order
  // 'ec_detr_fp:35' for ii = 1:reps
  b_i = ord_size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_regs, 2, &sc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_lin, 2, &tc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_fv, 1, &ic_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_fv1, 1, &ic_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_z, 1, &uc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_boolean_T#" MW_AT_LINE);
  emxInit_boolean_T(&b_cpu_x, 1, &ac_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_x, 1, &bc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_x, 1, &bc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_r, 2, &vc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_y, 2, &wc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_V, 2, &ic_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_D, 2, &ic_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_V, 2, &xc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_d, 1, &yc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &ad_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_y, 2, &bd_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_A, 2, &mc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_y, 2, &cd_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_y, 1, &dd_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_fv2, 1, &ed_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_d, 1, &qc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&e_cpu_y, 1, &fd_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_absdiff, 1, &gd_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_r, 2, &hd_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_r1, 2, &id_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_regs, 2, &bc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_r2, 2, &ec_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_V, 2, &jc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_r3, 2, &ec_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f_cpu_y, 1, &oc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_r4, 2, &lc_emlrtRTEI, true);
  profileLoopStart("ec_detr_fp_loop_0", __LINE__, (b_i - 1) + 1, "1957");
  for (int32_T ii{0}; ii < b_i; ii++) {
    int32_T k;
    int32_T nx;
    int32_T nz;
    boolean_T regs_needsGpuEnsureCapacity;
    boolean_T regs_outdatedOnCpu;
    boolean_T validLaunchParams;
    // 'ec_detr_fp:36' iOrd=ord(ii);
    // 'ec_detr_fp:36' iThr=thr(ii);
    // 'ec_detr_fp:36' iItr=itr(ii);
    //  copy
    // 'ec_detr_fp:37' regs = regsFromBasis_lfn(nFrames,iOrd);
    // %%%%%%%%%%%%%%%%%%%%%%%%%% SUBFUNCTIONS
    // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //  Get regressors from polynomial function %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    // 'ec_detr_fp:58' regs = coder.nullcopy(zeros(nFrames,iOrd,'like',iOrd));
    regs_outdatedOnCpu = false;
    i = cpu_regs->size[0] * cpu_regs->size[1];
    cpu_regs->size[0] = nFrames;
    cpu_regs->size[1] = static_cast<int32_T>(ord_data[ii]);
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_regs, i, &vb_emlrtRTEI);
    regs_needsGpuEnsureCapacity = true;
    // 'ec_detr_fp:59' lin = linspace(-1,1,nFrames);
    cpu_lin->size[0] = 1;
    cpu_lin->size[1] = nFrames;
    lin_needsGpuEnsureCapacity = true;
    if (nFrames >= 1) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_lin, &gpu_lin, true);
      lin_needsGpuEnsureCapacity = false;
      nvtxMarkA("#ec_detr_fp_kernel1#" MW_AT_LINE "#I1958");
      ec_detr_fp_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(nFrames,
                                                                  gpu_lin);
      if (cpu_lin->size[1] >= 2) {
        nvtxMarkA("#ec_detr_fp_kernel2#" MW_AT_LINE "#I1959");
        ec_detr_fp_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_lin);
        if (cpu_lin->size[1] >= 3) {
          real_T d2scaled;
          d2scaled = 1.0 / (static_cast<real_T>(cpu_lin->size[1]) - 1.0);
          nz = cpu_lin->size[1];
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nz - 3), &grid, &block,
                                  2147483647U);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel3#" MW_AT_LINE "#I1960");
            ec_detr_fp_kernel3<<<grid, block>>>(d2scaled, nz - 3, gpu_lin,
                                                cpu_lin->size[1U]);
          }
          if ((static_cast<uint32_T>(cpu_lin->size[1]) & 1U) == 1U) {
            nvtxMarkA("#ec_detr_fp_kernel4#" MW_AT_LINE "#I1961");
            ec_detr_fp_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                gpu_lin, cpu_lin->size[1U]);
          }
        }
      }
    }
    // 'ec_detr_fp:60' for k = 1:iOrd
    nz = static_cast<int32_T>(ord_data[ii]);
    profileLoopStart("ec_detr_fp_loop_1", __LINE__, (nz - 1) + 1, "1962");
    for (k = 0; k < nz; k++) {
      // 'ec_detr_fp:61' regs(:,k) = lin.^k;
      nx = cpu_lin->size[1];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      if (lin_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_lin, &gpu_lin, true);
      }
      lin_needsGpuEnsureCapacity = false;
      if (regs_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_regs, &gpu_regs, true);
      }
      regs_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_detr_fp_kernel5#" MW_AT_LINE "#I1963");
        ec_detr_fp_kernel5<<<grid, block>>>(
            k, gpu_lin, static_cast<int32_T>(static_cast<real32_T>(k) + 1.0F),
            nx - 1, gpu_regs, cpu_regs->size[0U]);
      }
      regs_outdatedOnCpu = true;
    }
    profileLoopEnd();
    //  Get regressors from basis function
    //  Detrend per chan
    // 'ec_detr_fp:40' for ch = 1:nChs
    k = cpu_x->size[1];
    if (k - 1 >= 0) {
      i5 = static_cast<int32_T>(itr_data[ii]);
    }
    profileLoopStart("ec_detr_fp_loop_2", __LINE__, (k - 1) + 1, "1964");
    for (int32_T ch{0}; ch < k; ch++) {
      int32_T vlen;
      boolean_T fv1_outdatedOnCpu;
      boolean_T fv_outdatedOnCpu;
      boolean_T z_outdatedOnCpu;
      // 'ec_detr_fp:41' [x(:,ch),w(:,ch)] =
      // detrend_lfn(x(:,ch),w(:,ch),regs,iThr,iItr);
      i = cpu_fv->size[0];
      cpu_fv->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_fv, i, &wb_emlrtRTEI);
      nx = cpu_x->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      if (x_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !fv2_outdatedOnGpu);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
      if (fv2_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_detr_fp_kernel6#" MW_AT_LINE "#I1965");
        ec_detr_fp_kernel6<<<grid, block>>>(ch, gpu_x, nx, gpu_fv,
                                            cpu_x->size[0U]);
      }
      fv_outdatedOnCpu = true;
      i = cpu_fv1->size[0];
      cpu_fv1->size[0] = cpu_w->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_fv1, i, &xb_emlrtRTEI);
      nx = cpu_w->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      if (y_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_fv1, &gpu_fv1, true);
      if (w_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_w, cpu_w);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_detr_fp_kernel7#" MW_AT_LINE "#I1966");
        ec_detr_fp_kernel7<<<grid, block>>>(ch, gpu_w, nx, gpu_fv1,
                                            cpu_w->size[0U]);
      }
      fv1_outdatedOnCpu = true;
      //  Standard detrending (trend fit to entire data)
      //  %%%%%%%%%%%%%%%%%%%%%%%%% The data are fit to the basis using weighted
      //  least squares. The weight is updated by setting samples for which the
      //  residual is greater than 'thresh' times its std to zero, and the fit
      //  is repeated at most 'niter'-1 times.
      //
      //  The choice of order (and basis) determines what complexity of the
      //  trend that can be removed.  It may be useful to first detrend with a
      //  low order to avoid fitting outliers, and then increase the order.
      //
      //  The tricky bit is to ensure that weighted means are removed before
      //  calculating the regression (see nt_regw)
      //  Detrend per polynomial order
      // 'ec_detr_fp:80' z = coder.nullcopy(x);
      z_outdatedOnCpu = false;
      lin_needsGpuEnsureCapacity = false;
      i = cpu_z->size[0];
      cpu_z->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_z, i, &yb_emlrtRTEI);
      w_outdatedOnGpu = true;
      // 'ec_detr_fp:81' for ii = 1:iItr
      profileLoopStart("ec_detr_fp_loop_3", __LINE__, (i5 - 1) + 1, "1967");
      for (int32_T b_ii{0}; b_ii < i5; b_ii++) {
        uint64_T numIters;
        real32_T b_mn;
        real32_T mn;
        real32_T xbar;
        //  Weighted regression
        // 'ec_detr_fp:83' z = regw_lfn(x,w,r);
        //  Weighted Regression
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        //   b: regression matrix (apply to r to approximate x)
        //   z: regression (r*b)
        //  Discard dimensions of r with eigenvalue lower than this
        // 'ec_detr_fp:102' thrPCA = 1e-7;
        // zeros(1,1,class(x));
        //  Save weighted mean
        // 'ec_detr_fp:105' mn = x - demean_lfn(x,w);
        //  Demean
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        // 'ec_detr_fp:124' mn = sum(x.*w,1) ./ (sum(w,1)+eps);
        if (cpu_fv->size[0] == cpu_fv1->size[0]) {
          i = c_cpu_x->size[0];
          c_cpu_x->size[0] = cpu_fv->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(c_cpu_x, i, &bc_emlrtRTEI);
          nx = cpu_fv->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(c_cpu_x, &b_gpu_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel8#" MW_AT_LINE "#I1968");
            ec_detr_fp_kernel8<<<grid, block>>>(gpu_fv1, gpu_fv, nx, b_gpu_x);
          }
          b_x_outdatedOnCpu = true;
        } else {
          if (b_x_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(c_cpu_x, &b_gpu_x);
          }
          if (fv_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
          }
          fv_outdatedOnCpu = false;
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          fv1_outdatedOnCpu = false;
          nvtxMarkA("#times#" MW_AT_LINE);
          times(c_cpu_x, cpu_fv, cpu_fv1);
          b_x_outdatedOnCpu = false;
        }
        vlen = c_cpu_x->size[0];
        if (c_cpu_x->size[0] == 0) {
          xbar = 0.0F;
        } else {
          if (b_x_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(c_cpu_x, &b_gpu_x);
          }
          b_x_outdatedOnCpu = false;
          xbar = c_cpu_x->data[0];
          profileLoopStart("ec_detr_fp_loop_4", __LINE__, (vlen - 2) + 1,
                           "1998");
          for (nx = 0; nx <= vlen - 2; nx++) {
            xbar += c_cpu_x->data[nx + 1];
          }
          profileLoopEnd();
        }
        vlen = cpu_fv1->size[0];
        if (cpu_fv1->size[0] == 0) {
          g_cpu_y = 0.0F;
        } else {
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          fv1_outdatedOnCpu = false;
          g_cpu_y = cpu_fv1->data[0];
          profileLoopStart("ec_detr_fp_loop_5", __LINE__, (vlen - 2) + 1,
                           "1997");
          for (nx = 0; nx <= vlen - 2; nx++) {
            g_cpu_y += cpu_fv1->data[nx + 1];
          }
          profileLoopEnd();
        }
        mn = xbar / (g_cpu_y + 2.22044605E-16F);
        // 'ec_detr_fp:125' x = x - mn;
        //  Fit weighted regression
        // 'ec_detr_fp:108' x = demean_lfn(x,w).* w;
        //  Demean
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        // 'ec_detr_fp:124' mn = sum(x.*w,1) ./ (sum(w,1)+eps);
        if (cpu_fv->size[0] == cpu_fv1->size[0]) {
          i = d_cpu_x->size[0];
          d_cpu_x->size[0] = cpu_fv->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(d_cpu_x, i, &bc_emlrtRTEI);
          nx = cpu_fv->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(d_cpu_x, &c_gpu_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel9#" MW_AT_LINE "#I1969");
            ec_detr_fp_kernel9<<<grid, block>>>(gpu_fv1, gpu_fv, nx, c_gpu_x);
          }
          c_x_outdatedOnCpu = true;
        } else {
          if (c_x_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(d_cpu_x, &c_gpu_x);
          }
          if (fv_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
          }
          fv_outdatedOnCpu = false;
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          fv1_outdatedOnCpu = false;
          nvtxMarkA("#times#" MW_AT_LINE);
          times(d_cpu_x, cpu_fv, cpu_fv1);
          c_x_outdatedOnCpu = false;
        }
        vlen = d_cpu_x->size[0];
        if (d_cpu_x->size[0] == 0) {
          xbar = 0.0F;
        } else {
          if (c_x_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(d_cpu_x, &c_gpu_x);
          }
          c_x_outdatedOnCpu = false;
          xbar = d_cpu_x->data[0];
          profileLoopStart("ec_detr_fp_loop_7", __LINE__, (vlen - 2) + 1,
                           "1996");
          for (nx = 0; nx <= vlen - 2; nx++) {
            xbar += d_cpu_x->data[nx + 1];
          }
          profileLoopEnd();
        }
        vlen = cpu_fv1->size[0];
        if (cpu_fv1->size[0] == 0) {
          g_cpu_y = 0.0F;
        } else {
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          fv1_outdatedOnCpu = false;
          g_cpu_y = cpu_fv1->data[0];
          profileLoopStart("ec_detr_fp_loop_8", __LINE__, (vlen - 2) + 1,
                           "1995");
          for (nx = 0; nx <= vlen - 2; nx++) {
            g_cpu_y += cpu_fv1->data[nx + 1];
          }
          profileLoopEnd();
        }
        b_mn = xbar / (g_cpu_y + 2.22044605E-16F);
        // 'ec_detr_fp:125' x = x - mn;
        // 'ec_detr_fp:109' r = demean_lfn(r,w);
        //  Demean
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        // 'ec_detr_fp:124' mn = sum(x.*w,1) ./ (sum(w,1)+eps);
        vlen = cpu_fv1->size[0];
        if (cpu_fv1->size[0] == 0) {
          g_cpu_y = 0.0F;
        } else {
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          fv1_outdatedOnCpu = false;
          g_cpu_y = cpu_fv1->data[0];
          profileLoopStart("ec_detr_fp_loop_9", __LINE__, (vlen - 2) + 1,
                           "1994");
          for (nx = 0; nx <= vlen - 2; nx++) {
            g_cpu_y += cpu_fv1->data[nx + 1];
          }
          profileLoopEnd();
        }
        // 'ec_detr_fp:125' x = x - mn;
        if (cpu_regs->size[0] == cpu_fv1->size[0]) {
          i = b_cpu_regs->size[0] * b_cpu_regs->size[1];
          b_cpu_regs->size[0] = cpu_regs->size[0];
          b_cpu_regs->size[1] = cpu_regs->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_cpu_regs, i, &bc_emlrtRTEI);
          nx = cpu_regs->size[1] - 1;
          nz = cpu_regs->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          numIters = computeNumIters(nx, nz);
          mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
          if (regs_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real32_T(cpu_regs, &gpu_regs, true);
          }
          regs_needsGpuEnsureCapacity = false;
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(b_cpu_regs, &b_gpu_regs, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel10#" MW_AT_LINE "#I1970");
            ec_detr_fp_kernel10<<<grid, block>>>(
                gpu_fv1, gpu_regs, nz, nx, b_gpu_regs, b_cpu_regs->size[0U],
                cpu_regs->size[0U]);
          }
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_regs, &b_gpu_regs);
          nvtxMarkA("#sum#" MW_AT_LINE);
          coder::sum(b_cpu_regs, b_cpu_r);
        } else {
          if (regs_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_regs, &gpu_regs);
          }
          regs_outdatedOnCpu = false;
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          fv1_outdatedOnCpu = false;
          nvtxMarkA("#binary_expand_op_4#" MW_AT_LINE);
          binary_expand_op_4(b_cpu_r, cpu_regs, cpu_fv1);
        }
        g_cpu_y += 2.22044605E-16F;
        i = cpu_r->size[0] * cpu_r->size[1];
        cpu_r->size[0] = cpu_regs->size[0];
        cpu_r->size[1] = cpu_regs->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_r, i, &cc_emlrtRTEI);
        nx = cpu_regs->size[1] - 1;
        nz = cpu_regs->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(nx, nz);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(b_cpu_r, &gpu_r, false);
        if (regs_needsGpuEnsureCapacity) {
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_regs, &gpu_regs, true);
        }
        regs_needsGpuEnsureCapacity = false;
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_r, &b_gpu_r, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_r, b_cpu_r);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel11#" MW_AT_LINE "#I1971");
          ec_detr_fp_kernel11<<<grid, block>>>(g_cpu_y, gpu_r, gpu_regs, nz, nx,
                                               b_gpu_r, cpu_r->size[0U],
                                               cpu_regs->size[0U]);
        }
        r_outdatedOnGpu = false;
        //  remove channel-specific-weighted mean from regressor
        // 'ec_detr_fp:110' rr = r.*w;
        if (cpu_r->size[0] == cpu_fv1->size[0]) {
          i = cpu_r1->size[0] * cpu_r1->size[1];
          cpu_r1->size[0] = cpu_r->size[0];
          cpu_r1->size[1] = cpu_r->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(cpu_r1, i, &dc_emlrtRTEI);
          nx = cpu_r->size[1] - 1;
          nz = cpu_r->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          numIters = computeNumIters(nx, nz);
          mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_r1, &gpu_r1, true);
          y_outdatedOnCpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel12#" MW_AT_LINE "#I1972");
            ec_detr_fp_kernel12<<<grid, block>>>(gpu_fv1, b_gpu_r, nz, nx,
                                                 gpu_r1, cpu_r1->size[0U],
                                                 cpu_r->size[0U]);
          }
          r1_outdatedOnGpu = false;
          r1_outdatedOnCpu = true;
        } else {
          if (r1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_r1, &gpu_r1);
          }
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_r, &b_gpu_r);
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          fv1_outdatedOnCpu = false;
          nvtxMarkA("#binary_expand_op_3#" MW_AT_LINE);
          binary_expand_op_3(cpu_r1, cpu_r, cpu_fv1);
          r1_outdatedOnCpu = false;
          r1_outdatedOnGpu = true;
          y_outdatedOnCpu = true;
        }
        // 'ec_detr_fp:111' [V,D] = eig(rr'*rr);
        i = cpu_r2->size[0] * cpu_r2->size[1];
        cpu_r2->size[0] = cpu_r1->size[1];
        cpu_r2->size[1] = cpu_r1->size[0];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_r2, i, &ec_emlrtRTEI);
        nx = cpu_r1->size[0] - 1;
        nz = cpu_r1->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(nx, nz);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        if (y_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_r1, &gpu_r1, !r1_outdatedOnGpu);
        }
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_r2, &gpu_r2, true);
        if (r1_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_r1, cpu_r1);
        }
        r1_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel13#" MW_AT_LINE "#I1973");
          ec_detr_fp_kernel13<<<grid, block>>>(
              gpu_r1, nz, nx, gpu_r2, cpu_r2->size[0U], cpu_r1->size[0U]);
        }
        lin_needsGpuEnsureCapacity = false;
        nvtxMarkA("#mtimes#" MW_AT_LINE);
        coder::internal::blas::mtimes(
            cpu_r2, &gpu_r2, &lin_needsGpuEnsureCapacity, cpu_r1, &gpu_r1,
            &r1_outdatedOnGpu, cpu_y, &y_outdatedOnCpu, &gpu_y,
            &y_outdatedOnGpu);
        if (y_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &gpu_y);
        }
        nvtxMarkA("#eig#" MW_AT_LINE);
        coder::eig(cpu_y, cpu_V, cpu_D);
        // 'ec_detr_fp:112' V = real(V);
        i = b_cpu_V->size[0] * b_cpu_V->size[1];
        b_cpu_V->size[0] = cpu_V->size[0];
        b_cpu_V->size[1] = cpu_V->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_cpu_V, i, &fc_emlrtRTEI);
        nx = cpu_V->size[0] * cpu_V->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(cpu_V, &gpu_V, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(b_cpu_V, &b_gpu_V, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_V, cpu_V);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel14#" MW_AT_LINE "#I1974");
          ec_detr_fp_kernel14<<<grid, block>>>(gpu_V, nx, b_gpu_V);
        }
        // 'ec_detr_fp:113' D = real(diag(D));
        if ((cpu_D->size[0] == 1) && (cpu_D->size[1] == 1)) {
          i = cpu_d->size[0];
          cpu_d->size[0] = 1;
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(cpu_d, i, &hc_emlrtRTEI);
          cpu_d->data[0] = cpu_D->data[0];
          fv2_outdatedOnCpu = false;
        } else {
          nz = cpu_D->size[0];
          vlen = cpu_D->size[1];
          if (nz <= vlen) {
            vlen = nz;
          }
          if (cpu_D->size[1] <= 0) {
            vlen = 0;
          }
          i = cpu_d->size[0];
          cpu_d->size[0] = vlen;
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(cpu_d, i, &gc_emlrtRTEI);
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(vlen - 1), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal32_T(cpu_D, &gpu_D, false);
          nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal32_T(cpu_d, &b_gpu_d, true);
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_D, cpu_D);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel27#" MW_AT_LINE "#I1993");
            ec_detr_fp_kernel27<<<grid, block>>>(gpu_D, vlen - 1, b_gpu_d,
                                                 cpu_D->size[0U]);
          }
          fv2_outdatedOnCpu = true;
        }
        // 'ec_detr_fp:114' V = V(:,D/max(D)>thrPCA);
        vlen = cpu_d->size[0];
        if (fv2_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal32_T(cpu_d, &b_gpu_d);
        }
        xbar = cpu_d->data[0].re;
        profileLoopStart("ec_detr_fp_loop_10", __LINE__, (vlen - 2) + 1,
                         "1975");
        for (i = 0; i <= vlen - 2; i++) {
          if (std::isnan(cpu_d->data[i + 1].re)) {
            fv2_outdatedOnCpu = false;
          } else if (std::isnan(xbar)) {
            fv2_outdatedOnCpu = true;
          } else {
            fv2_outdatedOnCpu = (xbar < cpu_d->data[i + 1].re);
          }
          if (fv2_outdatedOnCpu) {
            xbar = cpu_d->data[i + 1].re;
          }
        }
        profileLoopEnd();
        nx = cpu_d->size[0];
        nz = 0;
        profileLoopStart("ec_detr_fp_loop_11", __LINE__, (nx - 1) + 1, "1976");
        for (i = 0; i < nx; i++) {
          if (cpu_d->data[i].re / xbar > 1.0E-7) {
            nz++;
          }
        }
        profileLoopEnd();
        i = cpu_iv->size[0];
        cpu_iv->size[0] = nz;
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, i, &ic_emlrtRTEI);
        vlen = 0;
        profileLoopStart("ec_detr_fp_loop_12", __LINE__, (nx - 1) + 1, "1977");
        for (i = 0; i < nx; i++) {
          if (cpu_d->data[i].re / xbar > 1.0E-7) {
            cpu_iv->data[vlen] = i;
            iv_outdatedOnGpu = true;
            vlen++;
          }
        }
        profileLoopEnd();
        nz = b_cpu_V->size[0];
        c_cpu_V->size[0] = b_cpu_V->size[0];
        c_cpu_V->size[1] = cpu_iv->size[0];
        vlen = cpu_iv->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(vlen - 1, nz - 1);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, !iv_outdatedOnGpu);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(c_cpu_V, &c_gpu_V, true);
        if (iv_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_int32_T(&gpu_iv, cpu_iv);
        }
        iv_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel15#" MW_AT_LINE "#I1978");
          ec_detr_fp_kernel15<<<grid, block>>>(
              gpu_iv, b_gpu_V, nz - 1, vlen - 1, c_gpu_V, c_cpu_V->size[0U],
              b_cpu_V->size[0U]);
        }
        i = b_cpu_V->size[0] * b_cpu_V->size[1];
        b_cpu_V->size[0] = c_cpu_V->size[0];
        b_cpu_V->size[1] = c_cpu_V->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_cpu_V, i, &kc_emlrtRTEI);
        nx = c_cpu_V->size[0] * c_cpu_V->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(b_cpu_V, &b_gpu_V, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel16#" MW_AT_LINE "#I1979");
          ec_detr_fp_kernel16<<<grid, block>>>(c_gpu_V, nx, b_gpu_V);
        }
        fv2_outdatedOnGpu = false;
        //  discard weak dims
        // 'ec_detr_fp:115' rr = rr*V;
        w_outdatedOnGpu = false;
        i = cpu_r4->size[0] * cpu_r4->size[1];
        cpu_r4->size[0] = cpu_r1->size[0];
        cpu_r4->size[1] = cpu_r1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_r4, i, &lc_emlrtRTEI);
        vlen = cpu_r1->size[0] * cpu_r1->size[1] - 1;
        profileLoopStart("ec_detr_fp_loop_13", __LINE__, vlen + 1, "1980");
        for (i = 0; i <= vlen; i++) {
          if (r1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_r1, &gpu_r1);
          }
          r1_outdatedOnCpu = false;
          cpu_r4->data[i] = cpu_r1->data[i];
          w_outdatedOnGpu = true;
        }
        profileLoopEnd();
        nvtxMarkA("#mtimes#" MW_AT_LINE);
        coder::internal::blas::mtimes(cpu_r4, &gpu_r4, &w_outdatedOnGpu,
                                      b_cpu_V, &b_gpu_V, &fv2_outdatedOnGpu,
                                      cpu_r1, &r1_outdatedOnCpu, &gpu_r1,
                                      &r1_outdatedOnGpu);
        y_outdatedOnCpu = true;
        // 'ec_detr_fp:116' b = (x'*rr) / (rr'*rr);
        if (cpu_fv->size[0] == cpu_fv1->size[0]) {
          i = cpu_A->size[0] * cpu_A->size[1];
          cpu_A->size[0] = 1;
          cpu_A->size[1] = cpu_fv->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(cpu_A, i, &mc_emlrtRTEI);
          nx = cpu_fv->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_A, &gpu_A, true);
          w_outdatedOnGpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel17#" MW_AT_LINE "#I1981");
            ec_detr_fp_kernel17<<<grid, block>>>(gpu_fv1, b_mn, gpu_fv, nx,
                                                 gpu_A);
          }
          lin_needsGpuEnsureCapacity = false;
          A_outdatedOnCpu = true;
        } else {
          if (A_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_A, &gpu_A);
          }
          if (fv_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
          }
          fv_outdatedOnCpu = false;
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          fv1_outdatedOnCpu = false;
          nvtxMarkA("#binary_expand_op_2#" MW_AT_LINE);
          binary_expand_op_2(cpu_A, cpu_fv, b_mn, cpu_fv1);
          A_outdatedOnCpu = false;
          lin_needsGpuEnsureCapacity = true;
          w_outdatedOnGpu = true;
        }
        if ((cpu_A->size[1] == 0) || (cpu_r1->size[0] == 0) ||
            (cpu_r1->size[1] == 0)) {
          i = b_cpu_y->size[0] * b_cpu_y->size[1];
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = cpu_r1->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_cpu_y, i, &nc_emlrtRTEI);
          nz = cpu_r1->size[1];
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nz - 1), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(b_cpu_y, &b_gpu_y, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel18#" MW_AT_LINE "#I1982");
            ec_detr_fp_kernel18<<<grid, block>>>(nz - 1, b_gpu_y);
          }
          x_needsGpuEnsureCapacity = false;
          fv2_outdatedOnCpu = true;
        } else {
          i = b_cpu_y->size[0] * b_cpu_y->size[1];
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = cpu_r1->size[1];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_cpu_y, i, &c_emlrtRTEI);
          xbar = 1.0F;
          g_cpu_y = 0.0F;
          if (w_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real32_T(cpu_A, &gpu_A,
                                          !lin_needsGpuEnsureCapacity);
          }
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_r1, &gpu_r1, !r1_outdatedOnGpu);
          y_outdatedOnCpu = false;
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(b_cpu_y, &b_gpu_y, true);
          if (lin_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_A, cpu_A);
          }
          if (r1_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_r1, cpu_r1);
          }
          r1_outdatedOnGpu = false;
          hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 1,
                      cpu_r1->size[1], cpu_A->size[1], (float *)&xbar,
                      (float *)&gpu_A.data[0], 1, (float *)&gpu_r1.data[0],
                      cpu_A->size[1], (float *)&g_cpu_y,
                      (float *)&b_gpu_y.data[0], 1);
          x_needsGpuEnsureCapacity = false;
          fv2_outdatedOnCpu = true;
        }
        i = cpu_r3->size[0] * cpu_r3->size[1];
        cpu_r3->size[0] = cpu_r1->size[1];
        cpu_r3->size[1] = cpu_r1->size[0];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_r3, i, &ec_emlrtRTEI);
        nx = cpu_r1->size[0] - 1;
        nz = cpu_r1->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(nx, nz);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        if (y_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_r1, &gpu_r1, !r1_outdatedOnGpu);
        }
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_r3, &gpu_r3, true);
        if (r1_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_r1, cpu_r1);
        }
        r1_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel19#" MW_AT_LINE "#I1983");
          ec_detr_fp_kernel19<<<grid, block>>>(
              gpu_r1, nz, nx, gpu_r3, cpu_r3->size[0U], cpu_r1->size[0U]);
        }
        lin_needsGpuEnsureCapacity = false;
        nvtxMarkA("#mtimes#" MW_AT_LINE);
        coder::internal::blas::mtimes(
            cpu_r3, &gpu_r3, &lin_needsGpuEnsureCapacity, cpu_r1, &gpu_r1,
            &r1_outdatedOnGpu, c_cpu_y, &y_outdatedOnCpu, &c_gpu_y,
            &y_outdatedOnGpu);
        if (y_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(c_cpu_y, &c_gpu_y);
        }
        y_outdatedOnCpu = false;
        nvtxMarkA("#mrdiv#" MW_AT_LINE);
        coder::internal::mrdiv(b_cpu_y, &fv2_outdatedOnCpu, &b_gpu_y,
                               &x_needsGpuEnsureCapacity, c_cpu_y);
        // 'ec_detr_fp:117' z = r*(V*b') + mn;
        i = f_cpu_y->size[0];
        f_cpu_y->size[0] = b_cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(f_cpu_y, i, &oc_emlrtRTEI);
        nx = b_cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(b_cpu_y, &b_gpu_y,
                                      !x_needsGpuEnsureCapacity);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(f_cpu_y, &d_gpu_y, true);
        if (x_needsGpuEnsureCapacity) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&b_gpu_y, b_cpu_y);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel20#" MW_AT_LINE "#I1984");
          ec_detr_fp_kernel20<<<grid, block>>>(b_gpu_y, nx, d_gpu_y);
        }
        x_needsGpuEnsureCapacity = false;
        fv2_outdatedOnCpu = true;
        nvtxMarkA("#b_mtimes#" MW_AT_LINE);
        coder::internal::blas::b_mtimes(
            b_cpu_V, &b_gpu_V, &fv2_outdatedOnGpu, f_cpu_y, &d_gpu_y,
            &x_needsGpuEnsureCapacity, d_cpu_y, &y_outdatedOnCpu, &e_gpu_y,
            &y_outdatedOnGpu);
        nvtxMarkA("#b_mtimes#" MW_AT_LINE);
        coder::internal::blas::b_mtimes(cpu_r, &b_gpu_r, &r_outdatedOnGpu,
                                        d_cpu_y, &e_gpu_y, &y_outdatedOnGpu,
                                        cpu_fv2, &fv2_outdatedOnCpu, &gpu_fv2,
                                        &fv2_outdatedOnGpu);
        if (cpu_fv2->size[0] == cpu_fv->size[0]) {
          i = cpu_z->size[0];
          cpu_z->size[0] = cpu_fv2->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(cpu_z, i, &pc_emlrtRTEI);
          nx = cpu_fv2->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_fv2, &gpu_fv2, !fv2_outdatedOnGpu);
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z, true);
          w_outdatedOnGpu = false;
          if (fv2_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_fv2, cpu_fv2);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel21#" MW_AT_LINE "#I1985");
            ec_detr_fp_kernel21<<<grid, block>>>(mn, gpu_fv, gpu_fv2, nx,
                                                 gpu_z);
          }
          lin_needsGpuEnsureCapacity = false;
          z_outdatedOnCpu = true;
        } else {
          if (z_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_z, &gpu_z);
          }
          if (fv2_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv2, &gpu_fv2);
          }
          if (fv_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
          }
          fv_outdatedOnCpu = false;
          nvtxMarkA("#binary_expand_op_1#" MW_AT_LINE);
          binary_expand_op_1(cpu_z, cpu_fv2, cpu_fv, mn);
          z_outdatedOnCpu = false;
          lin_needsGpuEnsureCapacity = true;
          w_outdatedOnGpu = true;
        }
        //  Find outliers
        // 'ec_detr_fp:86' d = (x-z).*w;
        if (cpu_fv->size[0] == 1) {
          nx = cpu_z->size[0];
        } else {
          nx = cpu_fv->size[0];
        }
        if ((cpu_fv->size[0] == cpu_z->size[0]) && (nx == cpu_fv1->size[0])) {
          i = b_cpu_d->size[0];
          b_cpu_d->size[0] = cpu_fv->size[0];
          nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
          emxEnsureCapacity_real32_T(b_cpu_d, i, &qc_emlrtRTEI);
          nx = cpu_fv->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          if (w_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z,
                                          !lin_needsGpuEnsureCapacity);
          }
          w_outdatedOnGpu = false;
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(b_cpu_d, &gpu_d, true);
          fv2_outdatedOnGpu = false;
          if (lin_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_z, cpu_z);
          }
          lin_needsGpuEnsureCapacity = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel22#" MW_AT_LINE "#I1986");
            ec_detr_fp_kernel22<<<grid, block>>>(gpu_fv1, gpu_z, gpu_fv, nx,
                                                 gpu_d);
          }
          fv2_outdatedOnCpu = false;
          d_outdatedOnCpu = true;
        } else {
          if (d_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_d, &gpu_d);
          }
          if (fv_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
          }
          fv_outdatedOnCpu = false;
          if (z_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_z, &gpu_z);
          }
          z_outdatedOnCpu = false;
          if (fv1_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv1, &gpu_fv1);
          }
          nvtxMarkA("#binary_expand_op#" MW_AT_LINE);
          binary_expand_op(b_cpu_d, cpu_fv, cpu_z, cpu_fv1);
          d_outdatedOnCpu = false;
          fv2_outdatedOnCpu = true;
          fv2_outdatedOnGpu = true;
        }
        // 'ec_detr_fp:87' thrItr = iThr*std(d);
        vlen = b_cpu_d->size[0] - 1;
        if (b_cpu_d->size[0] == 0) {
          g_cpu_y = rtNaNF;
        } else if (b_cpu_d->size[0] == 1) {
          if (fv2_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real32_T(b_cpu_d, &gpu_d, !fv2_outdatedOnCpu);
          }
          fv2_outdatedOnGpu = false;
          if (fv2_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_d, b_cpu_d);
          }
          fv2_outdatedOnCpu = false;
          nvtxMarkA("#ec_detr_fp_kernel25#" MW_AT_LINE "#I1989");
          ec_detr_fp_kernel25<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_d,
                                                                       g_gpu_y);
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE "#I1990");
          checkCudaError(
              hipMemcpy(&g_cpu_y, g_gpu_y, 4UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        } else {
          if (d_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_d, &gpu_d);
          }
          d_outdatedOnCpu = false;
          xbar = b_cpu_d->data[0];
          profileLoopStart("ec_detr_fp_loop_14", __LINE__, (vlen - 1) + 1,
                           "1991");
          for (nx = 0; nx < vlen; nx++) {
            xbar += b_cpu_d->data[nx + 1];
          }
          profileLoopEnd();
          xbar /= static_cast<real32_T>(b_cpu_d->size[0]);
          cpu_absdiff->size[0] = b_cpu_d->size[0];
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(vlen), &grid, &block,
                                  2147483647U);
          if (fv2_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_real32_T(b_cpu_d, &gpu_d, !fv2_outdatedOnCpu);
          }
          fv2_outdatedOnGpu = false;
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_absdiff, &gpu_absdiff, true);
          if (fv2_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_d, b_cpu_d);
          }
          fv2_outdatedOnCpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_detr_fp_kernel26#" MW_AT_LINE "#I1992");
            ec_detr_fp_kernel26<<<grid, block>>>(xbar, gpu_d, vlen,
                                                 gpu_absdiff);
          }
          hipblasSnrm2(getCublasGlobalHandle(), b_cpu_d->size[0],
                      (float *)&gpu_absdiff.data[0], 1, (float *)&x);
          g_cpu_y = x / std::sqrt(static_cast<real32_T>(b_cpu_d->size[0] - 1));
        }
        // 'ec_detr_fp:88' w(abs(d)>thrItr) = 0;
        nx = b_cpu_d->size[0];
        e_cpu_y->size[0] = b_cpu_d->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        if (fv2_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(b_cpu_d, &gpu_d, !fv2_outdatedOnCpu);
        }
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(e_cpu_y, &f_gpu_y, true);
        if (fv2_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_d, b_cpu_d);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel23#" MW_AT_LINE "#I1987");
          ec_detr_fp_kernel23<<<grid, block>>>(gpu_d, nx - 1, f_gpu_y);
        }
        nx = e_cpu_y->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel24#" MW_AT_LINE "#I1988");
          ec_detr_fp_kernel24<<<grid, block>>>(thr_data[ii] * g_cpu_y, f_gpu_y,
                                               nx - 1, gpu_fv1);
        }
        fv1_outdatedOnCpu = true;
        //  update weights
      }
      profileLoopEnd();
      //  Remove trends
      // 'ec_detr_fp:92' x = x-z;
      nz = cpu_x->size[0];
      if (nz == cpu_z->size[0]) {
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nz - 1), &grid, &block,
                                2147483647U);
        if (w_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z,
                                        !lin_needsGpuEnsureCapacity);
        }
        x_needsGpuEnsureCapacity = false;
        if (lin_needsGpuEnsureCapacity) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_z, cpu_z);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_detr_fp_kernel28#" MW_AT_LINE "#I1999");
          ec_detr_fp_kernel28<<<grid, block>>>(gpu_z, ch, nz - 1, gpu_x,
                                               cpu_x->size[0U]);
        }
        fv2_outdatedOnGpu = false;
        x_outdatedOnCpu = true;
      } else {
        if (x_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_x, &gpu_x);
        }
        if (z_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_z, &gpu_z);
        }
        nvtxMarkA("#binary_expand_op_5#" MW_AT_LINE);
        binary_expand_op_5(cpu_x, ch, cpu_z);
        x_outdatedOnCpu = false;
        fv2_outdatedOnGpu = true;
        x_needsGpuEnsureCapacity = true;
      }
      nz = cpu_w->size[0];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nz - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_detr_fp_kernel29#" MW_AT_LINE "#I2000");
        ec_detr_fp_kernel29<<<grid, block>>>(gpu_fv1, ch, nz - 1, gpu_w,
                                             cpu_w->size[0U]);
      }
      fv2_outdatedOnCpu = true;
      //  Detrend
      // 'ec_detr_fp:42' olPct(ch,ii) = sum(~w(:,ch));
      i = b_cpu_x->size[0];
      b_cpu_x->size[0] = cpu_w->size[0];
      nvtxMarkA("#emxEnsureCapacity_boolean_T#" MW_AT_LINE);
      emxEnsureCapacity_boolean_T(b_cpu_x, i, &ac_emlrtRTEI);
      nx = cpu_w->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      y_outdatedOnCpu = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_boolean_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_boolean_T(b_cpu_x, &d_gpu_x, true);
      w_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_detr_fp_kernel30#" MW_AT_LINE "#I2001");
        ec_detr_fp_kernel30<<<grid, block>>>(ch, gpu_w, nx, d_gpu_x,
                                             cpu_w->size[0U]);
      }
      vlen = b_cpu_x->size[0];
      if (b_cpu_x->size[0] == 0) {
        nz = 0;
      } else {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_boolean_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_boolean_T(b_cpu_x, &d_gpu_x);
        nz = b_cpu_x->data[0];
        profileLoopStart("ec_detr_fp_loop_6", __LINE__, (vlen - 2) + 1, "2002");
        for (nx = 0; nx <= vlen - 2; nx++) {
          nz += b_cpu_x->data[nx + 1];
        }
        profileLoopEnd();
      }
      olPct->data[ch + olPct->size[0] * ii] = nz;
      //  Outlier percentages
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_r4);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f_cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_r3);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_V);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_r2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_regs);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_r1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_r);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_absdiff);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&e_cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_d);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_fv2);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_A);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_d);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_V);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_D);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_V);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_r);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_x);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_x);
  nvtxMarkA("#emxFree_boolean_T#" MW_AT_LINE);
  emxFree_boolean_T(&b_cpu_x);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_z);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_fv1);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_fv);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_lin);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_regs);
  //  Sum outliers across chans
  // 'ec_detr_fp:47' olPct = sum(olPct,1,"omitnan");
  nvtxMarkA("#sum#" MW_AT_LINE);
  coder::sum(olPct, olPct_data, olPct_size);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&olPct);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (x_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_x, &gpu_x);
  }
  if (fv2_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_w, &gpu_w);
  }
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_w);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_regs);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_lin);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_fv);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_fv1);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_z);
  nvtxMarkA("#gpuEmxFree_boolean_T#" MW_AT_LINE);
  gpuEmxFree_boolean_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_r);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_V);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_D);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_V);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&b_gpu_d);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_A);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&e_gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_fv2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_d);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&f_gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_absdiff);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_r1);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_regs);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_r2);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&c_gpu_V);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_r3);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_r4);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(g_gpu_y), __FILE__, __LINE__);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_detr_fp.cu)

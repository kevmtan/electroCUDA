//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mtimes.cu
//
// Code generation for function 'mtimes'
//

// Include files
#include "mtimes.h"
#include "ec_detr_fp_data.h"
#include "ec_detr_fp_emxutil.h"
#include "ec_detr_fp_mexutil.h"
#include "ec_detr_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Variable Definitions
static emlrtRTEInfo
    d_emlrtRTEI{
        140,      // lineNo
        5,        // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

// Function Definitions
//
//
namespace coder {
namespace internal {
namespace blas {
void b_mtimes(emxArray_real32_T *cpu_A, emxArray_real32_T *gpu_A,
              boolean_T *A_outdatedOnGpu, emxArray_real32_T *cpu_B,
              emxArray_real32_T *gpu_B, boolean_T *B_outdatedOnGpu,
              emxArray_real32_T *cpu_C, boolean_T *C_outdatedOnCpu,
              emxArray_real32_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  nvtxRangePushA("#fcn#b_mtimes#" MW_AT_LOCATION);
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0) || (cpu_B->size[0] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_C, i, &d_emlrtRTEI);
    profileLoopStart("b_mtimes_loop_0", __LINE__, (cpu_A->size[0] - 1) + 1, "");
    for (i = 0; i < cpu_A->size[0]; i++) {
      cpu_C->data[i] = 0.0F;
      *C_outdatedOnGpu = true;
    }
    profileLoopEnd();
  } else {
    int32_T i;
    real32_T alpha1;
    real32_T beta1;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_C, i, &c_emlrtRTEI);
    alpha1 = 1.0F;
    beta1 = 0.0F;
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_B, gpu_B, !*B_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    if (*B_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_B, cpu_B);
    }
    *B_outdatedOnGpu = false;
    hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                cpu_A->size[0], 1, cpu_A->size[1], (float *)&alpha1,
                (float *)&gpu_A->data[0], cpu_A->size[0],
                (float *)&gpu_B->data[0], cpu_A->size[1], (float *)&beta1,
                (float *)&gpu_C->data[0], cpu_A->size[0]);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
  nvtxRangePop();
}

//
//
void mtimes(emxArray_real32_T *cpu_A, emxArray_real32_T *gpu_A,
            boolean_T *A_outdatedOnGpu, emxArray_real32_T *cpu_B,
            emxArray_real32_T *gpu_B, boolean_T *B_outdatedOnGpu,
            emxArray_real32_T *cpu_C, boolean_T *C_outdatedOnCpu,
            emxArray_real32_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  nvtxRangePushA("#fcn#mtimes#" MW_AT_LOCATION);
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0) || (cpu_B->size[0] == 0) ||
      (cpu_B->size[1] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0] * cpu_C->size[1];
    cpu_C->size[0] = cpu_A->size[0];
    cpu_C->size[1] = cpu_B->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_C, i, &d_emlrtRTEI);
    profileLoopStart("mtimes_loop_0", __LINE__,
                     (cpu_A->size[0] * cpu_B->size[1] - 1) + 1, "");
    for (i = 0; i < cpu_A->size[0] * cpu_B->size[1]; i++) {
      cpu_C->data[i] = 0.0F;
      *C_outdatedOnGpu = true;
    }
    profileLoopEnd();
  } else {
    int32_T i;
    real32_T alpha1;
    real32_T beta1;
    i = cpu_C->size[0] * cpu_C->size[1];
    cpu_C->size[0] = cpu_A->size[0];
    cpu_C->size[1] = cpu_B->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_C, i, &c_emlrtRTEI);
    alpha1 = 1.0F;
    beta1 = 0.0F;
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_B, gpu_B, !*B_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    if (*B_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_B, cpu_B);
    }
    *B_outdatedOnGpu = false;
    hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                cpu_A->size[0], cpu_B->size[1], cpu_A->size[1],
                (float *)&alpha1, (float *)&gpu_A->data[0], cpu_A->size[0],
                (float *)&gpu_B->data[0], cpu_A->size[1], (float *)&beta1,
                (float *)&gpu_C->data[0], cpu_A->size[0]);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
  nvtxRangePop();
}

} // namespace blas
} // namespace internal
} // namespace coder

// End of code generation (mtimes.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// resample.cu
//
// Code generation for function 'resample'
//

// Include files
#include "resample.h"
#include "conv2.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "firls.h"
#include "kaiser.h"
#include "mod.h"
#include "rat.h"
#include "rt_nonfinite.h"
#include "uniformResampleKernel.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hip/hip_math_constants.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo hg_emlrtRTEI{
    161,                                                        // lineNo
    9,                                                          // colNo
    "resample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/resample.m" // pName
};

// Function Declarations
static __global__ void resample_kernel71(const emxArray_creal_T varargin_1,
                                         const int32_T b,
                                         emxArray_creal_T varargout_1);

static __global__ void resample_kernel72(const emxArray_real_T r,
                                         const int32_T b_r,
                                         emxArray_real_T b_dv);

static __global__ void resample_kernel73(const emxArray_real_T b_dv1,
                                         const emxArray_real_T b_dv,
                                         const int32_T c_dv,
                                         emxArray_real_T h1);

static __global__ void resample_kernel74(const real_T pqmax,
                                         const emxArray_real_T h1,
                                         const real_T opts_p,
                                         const int32_T b_h1,
                                         emxArray_real_T opts_filter);

static __global__ void resample_kernel75(const int32_T nZeroBegin,
                                         emxArray_real_T h);

static __global__ void resample_kernel76(const emxArray_uint32_T y,
                                         const real_T nZeroBegin,
                                         const int32_T b_y,
                                         emxArray_int32_T iv);

static __global__ void resample_kernel77(const emxArray_real_T opts_filter,
                                         const emxArray_int32_T iv,
                                         const int32_T b_iv, emxArray_real_T h);

static __global__ void resample_kernel78(const emxArray_creal_T varargout_1,
                                         const int32_T b_varargout_1,
                                         emxArray_creal_T b_y1);

static __global__ void resample_kernel79(const emxArray_creal_T b_y1,
                                         const int32_T b,
                                         emxArray_creal_T varargout_1);

static __global__ void resample_kernel80(const emxArray_creal_T w,
                                         const int32_T vlen, const int32_T b,
                                         emxArray_creal_T y);

static __global__ void resample_kernel81(emxArray_real_T y);

static __global__ void
resample_kernel82(const emxArray_creal_T y, const emxArray_real_T b_y,
                  const real_T opts_filterDelay, const int32_T ii,
                  const int32_T b, emxArray_creal_T b_y1, int32_T y1_dim0);

static __global__ void resample_kernel83(const int32_T b, emxArray_real_T y);

static __global__ void resample_kernel84(const emxArray_creal_T b_y1,
                                         const int32_T varargin_1,
                                         emxArray_creal_T varargout_1);

static __global__ void resample_kernel85(const emxArray_creal_T b_y1,
                                         const int32_T c_y1, const int32_T d_y1,
                                         emxArray_creal_T varargout_1,
                                         int32_T varargout_1_dim0,
                                         int32_T y1_dim0);

static __global__ void resample_kernel86(const emxArray_creal_T varargout_1,
                                         const int32_T b_varargout_1,
                                         emxArray_creal_T b_y1);

static __global__ void resample_kernel87(const int32_T b, emxArray_uint32_T y);

static __global__ void resample_kernel88(const emxArray_creal_T varargin_1,
                                         const int32_T b_varargin_1,
                                         emxArray_creal_T varargout_1);

static __global__ void
resample_kernel89(const emxArray_creal_T varargin_1, const int32_T b_varargin_1,
                  const int32_T c_varargin_1, emxArray_creal_T varargout_1,
                  int32_T varargout_1_dim0, int32_T varargin_1_dim0);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void resample_kernel71(
    const emxArray_creal_T varargin_1, const int32_T b,
    emxArray_creal_T varargout_1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    varargout_1.data[i1] = varargin_1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel72(
    const emxArray_real_T r, const int32_T b_r, emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_dv.data[i1] = r.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel73(
    const emxArray_real_T b_dv1, const emxArray_real_T b_dv, const int32_T c_dv,
    emxArray_real_T h1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h1.data[i1] = b_dv.data[i1] * b_dv1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel74(
    const real_T pqmax, const emxArray_real_T h1, const real_T opts_p,
    const int32_T b_h1, emxArray_real_T opts_filter)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_h1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_filter.data[i1] = opts_p * h1.data[i1] / pqmax;
  }
}

static __global__
    __launch_bounds__(1024, 1) void resample_kernel75(const int32_T nZeroBegin,
                                                      emxArray_real_T h)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nZeroBegin);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h.data[i1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel76(
    const emxArray_uint32_T y, const real_T nZeroBegin, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    iv.data[i1] =
        static_cast<int32_T>(nZeroBegin + static_cast<real_T>(y.data[i1]));
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel77(
    const emxArray_real_T opts_filter, const emxArray_int32_T iv,
    const int32_T b_iv, emxArray_real_T h)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_iv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h.data[iv.data[i1] - 1] = opts_filter.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel78(
    const emxArray_creal_T varargout_1, const int32_T b_varargout_1,
    emxArray_creal_T b_y1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_varargout_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1] = varargout_1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel79(
    const emxArray_creal_T b_y1, const int32_T b, emxArray_creal_T varargout_1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    varargout_1.data[i1] = b_y1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel80(
    const emxArray_creal_T w, const int32_T vlen, const int32_T b,
    emxArray_creal_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = w.data[vlen * i1];
  }
}

static __global__ __launch_bounds__(32,
                                    1) void resample_kernel81(emxArray_real_T y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    y.data[0] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel82(
    const emxArray_creal_T y, const emxArray_real_T b_y,
    const real_T opts_filterDelay, const int32_T ii, const int32_T b,
    emxArray_creal_T b_y1, int32_T y1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1 + y1_dim0 * ii] =
        y.data[static_cast<int32_T>(opts_filterDelay + b_y.data[i1]) - 1];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void resample_kernel83(const int32_T b,
                                                              emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<real_T>(i1) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel84(
    const emxArray_creal_T b_y1, const int32_T varargin_1,
    emxArray_creal_T varargout_1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(varargin_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    varargout_1.data[i1] = b_y1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel85(
    const emxArray_creal_T b_y1, const int32_T c_y1, const int32_T d_y1,
    emxArray_creal_T varargout_1, int32_T varargout_1_dim0, int32_T y1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(d_y1) + 1UL) *
                (static_cast<uint64_T>(c_y1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(c_y1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(c_y1) + 1UL));
    varargout_1.data[k + varargout_1_dim0 * i1] = b_y1.data[i1 + y1_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel86(
    const emxArray_creal_T varargout_1, const int32_T b_varargout_1,
    emxArray_creal_T b_y1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_varargout_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1] = varargout_1.data[i1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void resample_kernel87(const int32_T b,
                                                      emxArray_uint32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<uint32_T>(i1) + 1U;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel88(
    const emxArray_creal_T varargin_1, const int32_T b_varargin_1,
    emxArray_creal_T varargout_1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_varargin_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    varargout_1.data[i1] = varargin_1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel89(
    const emxArray_creal_T varargin_1, const int32_T b_varargin_1,
    const int32_T c_varargin_1, emxArray_creal_T varargout_1,
    int32_T varargout_1_dim0, int32_T varargin_1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_varargin_1) + 1UL) *
                (static_cast<uint64_T>(b_varargin_1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_varargin_1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_varargin_1) + 1UL));
    varargout_1.data[k + varargout_1_dim0 * i1] =
        varargin_1.data[i1 + varargin_1_dim0 * k];
  }
}

//
//
namespace coder {
void resample(emxArray_creal_T *cpu_varargin_1,
              emxArray_creal_T *gpu_varargin_1,
              boolean_T *varargin_1_outdatedOnGpu, real_T varargin_2,
              real_T varargin_3, emxArray_creal_T *cpu_varargout_1,
              boolean_T *varargout_1_outdatedOnCpu,
              emxArray_creal_T *gpu_varargout_1,
              boolean_T *varargout_1_outdatedOnGpu)
{
  static const int32_T iv1[2]{1, 7};
  static const int32_T iv2[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_y;
  emxArray_creal_T b_yCol;
  emxArray_creal_T gpu_w;
  emxArray_creal_T gpu_y1;
  emxArray_creal_T *b_cpu_y;
  emxArray_creal_T *cpu_w;
  emxArray_creal_T *cpu_y1;
  emxArray_creal_T *yCol;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T *cpu_iv;
  emxArray_real_T c_gpu_y;
  emxArray_real_T gpu_dv;
  emxArray_real_T gpu_dv1;
  emxArray_real_T gpu_h;
  emxArray_real_T gpu_h1;
  emxArray_real_T gpu_opts_filter;
  emxArray_real_T gpu_r;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_dv;
  emxArray_real_T *cpu_dv1;
  emxArray_real_T *cpu_h;
  emxArray_real_T *cpu_h1;
  emxArray_real_T *cpu_opts_filter;
  emxArray_real_T *cpu_r;
  emxArray_uint32_T gpu_y;
  emxArray_uint32_T *cpu_y;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T b_dv[4];
  real_T Ly;
  real_T fc;
  real_T nZeroBegin;
  real_T opts_filterDelay;
  real_T opts_p;
  real_T opts_q;
  real_T pqmax;
  int32_T varargout_1[1];
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T opts_dim;
  int32_T u1;
  int32_T varargin_1;
  int32_T vlen;
  uint32_T outSize_idx_1;
  char_T unusedExpr[23];
  boolean_T b_y_needsGpuEnsureCapacity;
  boolean_T h_outdatedOnCpu;
  boolean_T h_outdatedOnGpu;
  boolean_T opts_isRowVectorInput;
  boolean_T validLaunchParams;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T y1_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  gpuEmxReset_real_T(&c_gpu_y);
  gpuEmxReset_creal_T(&b_gpu_y);
  gpuEmxReset_creal_T(&gpu_w);
  gpuEmxReset_creal_T(&gpu_y1);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_uint32_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_h);
  gpuEmxReset_real_T(&gpu_opts_filter);
  gpuEmxReset_real_T(&gpu_h1);
  gpuEmxReset_real_T(&gpu_dv1);
  gpuEmxReset_real_T(&gpu_dv);
  gpuEmxReset_real_T(&gpu_r);
  w_outdatedOnGpu = false;
  b_y_needsGpuEnsureCapacity = true;
  y_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_varargin_1->size[0] == 1) || (cpu_varargin_1->size[1] == 1)) {
    opts_dim = 1;
  } else {
    opts_dim = 2;
    if (cpu_varargin_1->size[0] != 1) {
      opts_dim = 1;
    }
  }
  opts_isRowVectorInput = (cpu_varargin_1->size[0] == 1);
  if (opts_dim == 1) {
    if ((cpu_varargin_1->size[0] == 1) || (cpu_varargin_1->size[1] == 1)) {
      if ((cpu_varargin_1->size[0] == 0) || (cpu_varargin_1->size[1] == 0)) {
        u1 = 0;
      } else {
        vlen = cpu_varargin_1->size[0];
        u1 = cpu_varargin_1->size[1];
        if (vlen >= u1) {
          u1 = vlen;
        }
      }
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = u1;
      cpu_varargout_1->size[1] = 1;
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &g_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(u1 - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_creal_T(cpu_varargin_1, gpu_varargin_1,
                                   !*varargin_1_outdatedOnGpu);
      gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
      if (*varargin_1_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(gpu_varargin_1, cpu_varargin_1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel71<<<grid, block>>>(*gpu_varargin_1, u1 - 1,
                                           *gpu_varargout_1);
      }
      *varargout_1_outdatedOnGpu = false;
      *varargout_1_outdatedOnCpu = true;
    } else {
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = cpu_varargin_1->size[0];
      cpu_varargout_1->size[1] = cpu_varargin_1->size[1];
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &g_emlrtRTEI);
      i = cpu_varargin_1->size[0] * cpu_varargin_1->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal_T(cpu_varargin_1, gpu_varargin_1,
                                   !*varargin_1_outdatedOnGpu);
      gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
      if (*varargin_1_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(gpu_varargin_1, cpu_varargin_1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel88<<<grid, block>>>(*gpu_varargin_1, i,
                                           *gpu_varargout_1);
      }
      *varargout_1_outdatedOnGpu = false;
      *varargout_1_outdatedOnCpu = true;
    }
  } else {
    i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
    cpu_varargout_1->size[0] = cpu_varargin_1->size[1];
    cpu_varargout_1->size[1] = cpu_varargin_1->size[0];
    emxEnsureCapacity_creal_T(cpu_varargout_1, i, &g_emlrtRTEI);
    i = cpu_varargin_1->size[0] - 1;
    varargin_1 = cpu_varargin_1->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i, varargin_1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_creal_T(cpu_varargin_1, gpu_varargin_1,
                                 !*varargin_1_outdatedOnGpu);
    gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
    if (*varargin_1_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_varargin_1, cpu_varargin_1);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      resample_kernel89<<<grid, block>>>(
          *gpu_varargin_1, varargin_1, i, *gpu_varargout_1,
          cpu_varargout_1->size[0U], cpu_varargin_1->size[0U]);
    }
    *varargout_1_outdatedOnGpu = false;
    *varargout_1_outdatedOnCpu = true;
  }
  opts_p = rat(varargin_2 / varargin_3, &opts_q);
  pqmax = std::fmax(opts_p, opts_q);
  fc = 0.5 / pqmax;
  pqmax = 20.0 * pqmax + 1.0;
  b_dv[0] = 0.0;
  b_dv[1] = 2.0 * fc;
  b_dv[2] = 2.0 * fc;
  b_dv[3] = 1.0;
  emxInit_real_T(&cpu_r, 2, &k_emlrtRTEI, true);
  firls(pqmax - 1.0, b_dv, cpu_r);
  emxInit_real_T(&cpu_dv, 1, &k_emlrtRTEI, true);
  i = cpu_dv->size[0];
  cpu_dv->size[0] = cpu_r->size[1];
  emxEnsureCapacity_real_T(cpu_dv, i, &k_emlrtRTEI);
  i = cpu_r->size[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_r, &gpu_r, false);
  gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
  gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, cpu_r);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel72<<<grid, block>>>(gpu_r, i, gpu_dv);
  }
  emxFree_real_T(&cpu_r);
  emxInit_real_T(&cpu_dv1, 1, &k_emlrtRTEI, true);
  kaiser(pqmax, cpu_dv1);
  emxInit_real_T(&cpu_h1, 1, &l_emlrtRTEI, true);
  i = cpu_h1->size[0];
  cpu_h1->size[0] = cpu_dv->size[0];
  emxEnsureCapacity_real_T(cpu_h1, i, &l_emlrtRTEI);
  varargin_1 = cpu_dv->size[0] - 1;
  mwGetLaunchParameters1D(computeNumIters(varargin_1), &grid, &block,
                          2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, false);
  gpuEmxEnsureCapacity_real_T(cpu_h1, &gpu_h1, true);
  w_outdatedOnCpu = false;
  gpuEmxMemcpyCpuToGpu_real_T(&gpu_dv1, cpu_dv1);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel73<<<grid, block>>>(gpu_dv1, gpu_dv, varargin_1, gpu_h1);
  }
  emxFree_real_T(&cpu_dv1);
  emxFree_real_T(&cpu_dv);
  vlen = cpu_h1->size[0];
  if (cpu_h1->size[0] == 0) {
    pqmax = 0.0;
  } else {
    gpuEmxMemcpyGpuToCpu_real_T(cpu_h1, &gpu_h1);
    pqmax = cpu_h1->data[0];
    w_outdatedOnCpu = true;
    for (varargin_1 = 0; varargin_1 <= vlen - 2; varargin_1++) {
      pqmax += cpu_h1->data[varargin_1 + 1];
    }
  }
  emxInit_real_T(&cpu_opts_filter, 1, &g_emlrtRTEI, true);
  i = cpu_opts_filter->size[0];
  cpu_opts_filter->size[0] = cpu_h1->size[0];
  emxEnsureCapacity_real_T(cpu_opts_filter, i, &g_emlrtRTEI);
  i = cpu_h1->size[0] - 1;
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  if (w_outdatedOnCpu) {
    gpuEmxEnsureCapacity_real_T(cpu_h1, &gpu_h1, true);
  }
  gpuEmxEnsureCapacity_real_T(cpu_opts_filter, &gpu_opts_filter, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel74<<<grid, block>>>(pqmax, gpu_h1, opts_p, i,
                                       gpu_opts_filter);
  }
  emxFree_real_T(&cpu_h1);
  pqmax = (static_cast<real_T>(cpu_opts_filter->size[0]) - 1.0) / 2.0;
  nZeroBegin = std::floor(opts_q - b_mod(pqmax, opts_q));
  opts_filterDelay = std::floor(std::ceil(pqmax + nZeroBegin) / opts_q);
  pqmax = static_cast<real_T>(cpu_opts_filter->size[0]) + nZeroBegin;
  fc = 0.0;
  vlen = cpu_varargout_1->size[0];
  while (std::ceil((((static_cast<real_T>(vlen) - 1.0) * opts_p + pqmax) + fc) /
                   opts_q) -
             opts_filterDelay <
         std::ceil(static_cast<real_T>(vlen) * opts_p / opts_q)) {
    fc++;
  }
  emxInit_real_T(&cpu_h, 1, &m_emlrtRTEI, true);
  i = cpu_h->size[0];
  cpu_h->size[0] = static_cast<int32_T>(
      (nZeroBegin + static_cast<real_T>(cpu_opts_filter->size[0])) + fc);
  emxEnsureCapacity_real_T(cpu_h, i, &m_emlrtRTEI);
  i = static_cast<int32_T>(
          (nZeroBegin + static_cast<real_T>(cpu_opts_filter->size[0])) + fc) -
      1;
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_h, &gpu_h, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel75<<<grid, block>>>(i, gpu_h);
  }
  emxInit_uint32_T(&cpu_y, 2, &ab_emlrtRTEI, true);
  if (cpu_opts_filter->size[0] < 1) {
    cpu_y->size[0] = 1;
    cpu_y->size[1] = 0;
  } else {
    outSize_idx_1 = static_cast<uint32_T>(cpu_opts_filter->size[0]);
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_opts_filter->size[0];
    emxEnsureCapacity_uint32_T(cpu_y, i, &n_emlrtRTEI);
    mwGetLaunchParameters1D(
        computeNumIters(static_cast<int32_T>(outSize_idx_1) - 1), &grid, &block,
        2147483647U);
    gpuEmxEnsureCapacity_uint32_T(cpu_y, &gpu_y, true);
    y_needsGpuEnsureCapacity = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      resample_kernel87<<<grid, block>>>(
          static_cast<int32_T>(outSize_idx_1) - 1, gpu_y);
    }
  }
  emxInit_int32_T(&cpu_iv, 2, &bb_emlrtRTEI, true);
  i = cpu_iv->size[0] * cpu_iv->size[1];
  cpu_iv->size[0] = 1;
  cpu_iv->size[1] = cpu_y->size[1];
  emxEnsureCapacity_int32_T(cpu_iv, i, &o_emlrtRTEI);
  i = cpu_y->size[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  if (y_needsGpuEnsureCapacity) {
    gpuEmxEnsureCapacity_uint32_T(cpu_y, &gpu_y, true);
  }
  gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel76<<<grid, block>>>(gpu_y, nZeroBegin, i, gpu_iv);
  }
  emxFree_uint32_T(&cpu_y);
  i = cpu_iv->size[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    resample_kernel77<<<grid, block>>>(gpu_opts_filter, gpu_iv, i, gpu_h);
  }
  h_outdatedOnGpu = false;
  h_outdatedOnCpu = true;
  emxFree_int32_T(&cpu_iv);
  emxFree_real_T(&cpu_opts_filter);
  emxInit_creal_T(&cpu_y1, 2, &cb_emlrtRTEI, true);
  if (opts_dim == 1) {
    if (opts_isRowVectorInput) {
      i = cpu_y1->size[0] * cpu_y1->size[1];
      cpu_y1->size[0] = cpu_varargout_1->size[0];
      cpu_y1->size[1] = cpu_varargout_1->size[1];
      emxEnsureCapacity_creal_T(cpu_y1, i, &r_emlrtRTEI);
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal_T(cpu_y1, &gpu_y1, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel78<<<grid, block>>>(*gpu_varargout_1, i, gpu_y1);
      }
      y1_outdatedOnGpu = false;
      w_outdatedOnCpu = true;
      b_signal::internal::resample::b_uniformResampleAlongFirstDim(
          cpu_y1, &w_outdatedOnCpu, &gpu_y1, &y1_outdatedOnGpu, opts_p, opts_q,
          cpu_h, &h_outdatedOnCpu, &gpu_h, &h_outdatedOnGpu, opts_filterDelay);
      if ((cpu_y1->size[0] == 0) || (cpu_y1->size[1] == 0)) {
        u1 = 0;
      } else {
        vlen = cpu_y1->size[0];
        u1 = cpu_y1->size[1];
        if (vlen >= u1) {
          u1 = vlen;
        }
      }
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = 1;
      cpu_varargout_1->size[1] = u1;
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &hg_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(u1 - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_creal_T(cpu_y1, &gpu_y1, !y1_outdatedOnGpu);
      gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
      if (y1_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_y1, cpu_y1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel79<<<grid, block>>>(gpu_y1, u1 - 1, *gpu_varargout_1);
      }
    } else {
      b_signal::internal::resample::b_uniformResampleAlongFirstDim(
          cpu_varargout_1, varargout_1_outdatedOnCpu, gpu_varargout_1,
          varargout_1_outdatedOnGpu, opts_p, opts_q, cpu_h, &h_outdatedOnCpu,
          &gpu_h, &h_outdatedOnGpu, opts_filterDelay);
    }
  } else {
    if (cpu_varargout_1->size[0] == 1) {
      pqmax = std::ceil(opts_p / opts_q);
      y1_outdatedOnGpu = false;
      i = cpu_y1->size[0] * cpu_y1->size[1];
      cpu_y1->size[0] = static_cast<int32_T>(pqmax);
      cpu_y1->size[1] = cpu_varargout_1->size[1];
      emxEnsureCapacity_creal_T(cpu_y1, i, &s_emlrtRTEI);
      opts_isRowVectorInput = true;
      opts_dim = cpu_varargout_1->size[1];
      if (cpu_varargout_1->size[1] - 1 >= 0) {
        if ((static_cast<int32_T>(opts_p) == 0) ||
            ((static_cast<int32_T>(opts_p) > 0) &&
             (static_cast<int32_T>(opts_p) < 1))) {
          i1 = 1;
          i2 = 0;
        } else {
          i1 = static_cast<int32_T>(opts_p);
          i2 = static_cast<int32_T>(opts_p);
        }
        Ly = std::ceil((0.0 * opts_p + static_cast<real_T>(cpu_h->size[0])) /
                       opts_q);
      }
      emxInit_creal_T(&yCol, 1, &db_emlrtRTEI, true);
      emxInit_creal_T(&cpu_w, 1, &v_emlrtRTEI, true);
      emxInit_creal_T(&b_cpu_y, 1, &v_emlrtRTEI, true);
      emxInit_real_T(&c_cpu_y, 2, &eb_emlrtRTEI, true);
      for (int32_T ii{0}; ii < opts_dim; ii++) {
        if (!(opts_p - 1.0 >= 0.0)) {
          y = nullptr;
          m = emlrtCreateCharArray(2, &iv1[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
          emlrtAssign(&y, m);
          b_y = nullptr;
          m1 = emlrtCreateDoubleScalar(opts_p - 1.0);
          emlrtAssign(&b_y, m1);
          emlrt_marshallIn(b_sprintf(y, b_y, &emlrtMCI), "<output of sprintf>",
                           unusedExpr);
        }
        i = yCol->size[0];
        yCol->size[0] = static_cast<int32_T>(opts_p);
        emxEnsureCapacity_creal_T(yCol, i, &t_emlrtRTEI);
        for (varargin_1 = 0; varargin_1 < static_cast<int32_T>(opts_p);
             varargin_1++) {
          yCol->data[varargin_1].re = 0.0;
          yCol->data[varargin_1].im = 0.0;
        }
        vlen = div_s32(i2 - 1, i1) + 1;
        for (varargin_1 = 0; varargin_1 < vlen; varargin_1++) {
          if (*varargout_1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_creal_T(cpu_varargout_1, gpu_varargout_1);
          }
          *varargout_1_outdatedOnCpu = false;
          yCol->data[i1 * varargin_1] = cpu_varargout_1->data[ii];
        }
        b_yCol = *yCol;
        varargout_1[0] = static_cast<int32_T>(opts_p);
        b_yCol.size = &varargout_1[0];
        b_yCol.numDimensions = 1;
        d_conv2(&b_yCol, cpu_h, &h_outdatedOnCpu, &gpu_h, &h_outdatedOnGpu,
                cpu_w, &w_outdatedOnCpu, &gpu_w, &w_outdatedOnGpu);
        if (!(opts_q - 1.0 >= 0.0)) {
          c_y = nullptr;
          m2 = emlrtCreateCharArray(2, &iv2[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
          emlrtAssign(&c_y, m2);
          d_y = nullptr;
          m3 = emlrtCreateDoubleScalar(opts_q - 1.0);
          emlrtAssign(&d_y, m3);
          emlrt_marshallIn(b_sprintf(c_y, d_y, &emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        if (opts_q == 0.0) {
          vlen = 1;
          u1 = -1;
        } else {
          vlen = static_cast<int32_T>(opts_q);
          u1 = cpu_w->size[0] - 1;
        }
        i = b_cpu_y->size[0];
        b_cpu_y->size[0] = div_s32(u1, vlen) + 1;
        emxEnsureCapacity_creal_T(b_cpu_y, i, &v_emlrtRTEI);
        varargin_1 = u1 / vlen;
        mwGetLaunchParameters1D(computeNumIters(varargin_1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_creal_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
        gpuEmxEnsureCapacity_creal_T(b_cpu_y, &b_gpu_y, true);
        y_needsGpuEnsureCapacity = false;
        if (w_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_creal_T(&gpu_w, cpu_w);
        }
        w_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          resample_kernel80<<<grid, block>>>(gpu_w, vlen, varargin_1, b_gpu_y);
        }
        if (!(div_s32(u1, vlen) + 1 < Ly)) {
          i = b_cpu_y->size[0];
          if (Ly < 1.0) {
            b_cpu_y->size[0] = 0;
          } else {
            b_cpu_y->size[0] = static_cast<int32_T>(Ly);
          }
          emxEnsureCapacity_creal_T(b_cpu_y, i, &w_emlrtRTEI);
          y_needsGpuEnsureCapacity = true;
        }
        if (std::isnan(pqmax)) {
          i = c_cpu_y->size[0] * c_cpu_y->size[1];
          c_cpu_y->size[0] = 1;
          c_cpu_y->size[1] = 1;
          emxEnsureCapacity_real_T(c_cpu_y, i, &n_emlrtRTEI);
          gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
          b_y_needsGpuEnsureCapacity = false;
          resample_kernel81<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(c_gpu_y);
        } else if (pqmax < 1.0) {
          c_cpu_y->size[0] = 1;
          c_cpu_y->size[1] = 0;
        } else {
          i = c_cpu_y->size[0] * c_cpu_y->size[1];
          c_cpu_y->size[0] = 1;
          c_cpu_y->size[1] = static_cast<int32_T>(pqmax - 1.0) + 1;
          emxEnsureCapacity_real_T(c_cpu_y, i, &n_emlrtRTEI);
          mwGetLaunchParameters1D(
              computeNumIters(static_cast<int32_T>(pqmax - 1.0)), &grid, &block,
              2147483647U);
          gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
          b_y_needsGpuEnsureCapacity = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            resample_kernel83<<<grid, block>>>(
                static_cast<int32_T>(pqmax - 1.0), c_gpu_y);
          }
        }
        vlen = cpu_y1->size[0];
        mwGetLaunchParameters1D(computeNumIters(vlen - 1), &grid, &block,
                                2147483647U);
        if (y_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_creal_T(b_cpu_y, &b_gpu_y, true);
        }
        if (b_y_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
        }
        b_y_needsGpuEnsureCapacity = false;
        if (opts_isRowVectorInput) {
          gpuEmxEnsureCapacity_creal_T(cpu_y1, &gpu_y1, true);
        }
        opts_isRowVectorInput = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          resample_kernel82<<<grid, block>>>(b_gpu_y, c_gpu_y, opts_filterDelay,
                                             ii, vlen - 1, gpu_y1,
                                             cpu_y1->size[0U]);
        }
      }
      emxFree_real_T(&c_cpu_y);
      emxFree_creal_T(&b_cpu_y);
      emxFree_creal_T(&cpu_w);
      emxFree_creal_T(&yCol);
    } else {
      i = cpu_y1->size[0] * cpu_y1->size[1];
      cpu_y1->size[0] = cpu_varargout_1->size[0];
      cpu_y1->size[1] = cpu_varargout_1->size[1];
      emxEnsureCapacity_creal_T(cpu_y1, i, &p_emlrtRTEI);
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal_T(cpu_y1, &gpu_y1, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel86<<<grid, block>>>(*gpu_varargout_1, i, gpu_y1);
      }
      y1_outdatedOnGpu = false;
      w_outdatedOnCpu = true;
      b_signal::internal::resample::b_uniformResampleAlongFirstDim(
          cpu_y1, &w_outdatedOnCpu, &gpu_y1, &y1_outdatedOnGpu, opts_p, opts_q,
          cpu_h, &h_outdatedOnCpu, &gpu_h, &h_outdatedOnGpu, opts_filterDelay);
      opts_isRowVectorInput = true;
    }
    if ((cpu_varargout_1->size[0] == 1) || (cpu_varargout_1->size[1] == 1)) {
      outSize_idx_1 = static_cast<uint32_T>(cpu_y1->size[0]);
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = cpu_varargin_1->size[0];
      cpu_varargout_1->size[1] = cpu_y1->size[0];
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &hg_emlrtRTEI);
      i = cpu_varargin_1->size[0] * static_cast<int32_T>(outSize_idx_1) - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (opts_isRowVectorInput) {
        gpuEmxEnsureCapacity_creal_T(cpu_y1, &gpu_y1, !y1_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
      if (y1_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_y1, cpu_y1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel84<<<grid, block>>>(gpu_y1, i, *gpu_varargout_1);
      }
      *varargout_1_outdatedOnCpu = true;
    } else {
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = cpu_y1->size[1];
      cpu_varargout_1->size[1] = cpu_y1->size[0];
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &hg_emlrtRTEI);
      i = cpu_y1->size[0] - 1;
      varargin_1 = cpu_y1->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(i, varargin_1), &grid, &block,
                              2147483647U);
      if (opts_isRowVectorInput) {
        gpuEmxEnsureCapacity_creal_T(cpu_y1, &gpu_y1, !y1_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
      if (y1_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_y1, cpu_y1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        resample_kernel85<<<grid, block>>>(
            gpu_y1, varargin_1, i, *gpu_varargout_1, cpu_varargout_1->size[0U],
            cpu_y1->size[0U]);
      }
      *varargout_1_outdatedOnCpu = true;
    }
  }
  emxFree_creal_T(&cpu_y1);
  emxFree_real_T(&cpu_h);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_r);
  gpuEmxFree_real_T(&gpu_dv);
  gpuEmxFree_real_T(&gpu_dv1);
  gpuEmxFree_real_T(&gpu_h1);
  gpuEmxFree_real_T(&gpu_opts_filter);
  gpuEmxFree_real_T(&gpu_h);
  gpuEmxFree_uint32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_creal_T(&gpu_y1);
  gpuEmxFree_creal_T(&gpu_w);
  gpuEmxFree_creal_T(&b_gpu_y);
  gpuEmxFree_real_T(&c_gpu_y);
  *varargin_1_outdatedOnGpu = false;
}

} // namespace coder

// End of code generation (resample.cu)

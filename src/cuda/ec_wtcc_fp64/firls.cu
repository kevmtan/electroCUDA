#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// firls.cu
//
// Code generation for function 'firls'
//

// Include files
#include "firls.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "mldivide.h"
#include "rt_nonfinite.h"
#include <cfloat>
#include <cmath>

// Variable Definitions
static emlrtRTEInfo ae_emlrtRTEI{
    245,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo be_emlrtRTEI{
    243,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ce_emlrtRTEI{
    247,                                                     // lineNo
    5,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo de_emlrtRTEI{
    258,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ee_emlrtRTEI{
    274,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo fe_emlrtRTEI{
    275,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ge_emlrtRTEI{
    276,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo he_emlrtRTEI{
    277,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ie_emlrtRTEI{
    286,                                                     // lineNo
    5,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo je_emlrtRTEI{
    281,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ke_emlrtRTEI{
    341,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo le_emlrtRTEI{
    347,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo
    me_emlrtRTEI{
        76,                  // lineNo
        9,                   // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo ne_emlrtRTEI{
    1,                                                               // lineNo
    21,                                                              // colNo
    "flipud",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/flipud.m" // pName
};

static emlrtRTEInfo oe_emlrtRTEI{
    341,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo pe_emlrtRTEI{
    353,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo qe_emlrtRTEI{
    355,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo re_emlrtRTEI{
    253,                                                     // lineNo
    20,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo se_emlrtRTEI{
    274,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo te_emlrtRTEI{
    275,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ue_emlrtRTEI{
    276,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ve_emlrtRTEI{
    277,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo we_emlrtRTEI{
    1,                                                       // lineNo
    16,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo xe_emlrtRTEI{
    295,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ye_emlrtRTEI{
    355,                                                     // lineNo
    17,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo af_emlrtRTEI{
    297,                                                     // lineNo
    46,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo bf_emlrtRTEI{
    298,                                                     // lineNo
    39,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo gh_emlrtRTEI{
    297,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo hh_emlrtRTEI{
    297,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ih_emlrtRTEI{
    295,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo jh_emlrtRTEI{
    295,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

// Function Declarations
static void binary_expand_op_6(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3, real_T in4,
                               const emxArray_real_T *in5);

static void binary_expand_op_7(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3,
                               const emxArray_real_T *in4,
                               const emxArray_real_T *in5);

static real_T rt_remd_snf(real_T u0, real_T u1);

// Function Definitions
static void binary_expand_op_6(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3, real_T in4,
                               const emxArray_real_T *in5)
{
  emxArray_real_T *b_in1;
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&b_in1, 1, &gh_emlrtRTEI, true);
  i = b_in1->size[0];
  if (in3->size[0] == 1) {
    b_in1->size[0] = in1->size[0];
  } else {
    b_in1->size[0] = in3->size[0];
  }
  emxEnsureCapacity_real_T(b_in1, i, &gh_emlrtRTEI);
  stride_0_0 = (in1->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  if (in3->size[0] == 1) {
    b = in1->size[0];
  } else {
    b = in3->size[0];
  }
  for (i = 0; i < b; i++) {
    b_in1->data[i] =
        in1->data[i * stride_0_0] +
        (in2 * in3->data[i * stride_1_0] - in4 * in5->data[i * stride_1_0]);
  }
  i = in1->size[0];
  in1->size[0] = b_in1->size[0];
  emxEnsureCapacity_real_T(in1, i, &hh_emlrtRTEI);
  for (i = 0; i < b_in1->size[0]; i++) {
    in1->data[i] = b_in1->data[i];
  }
  emxFree_real_T(&b_in1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

static void binary_expand_op_7(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3,
                               const emxArray_real_T *in4,
                               const emxArray_real_T *in5)
{
  emxArray_real_T *b_in1;
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  int32_T stride_2_0;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&b_in1, 1, &ih_emlrtRTEI, true);
  i = b_in1->size[0];
  if (in5->size[0] == 1) {
    b = in3->size[0];
  } else {
    b = in5->size[0];
  }
  if (b == 1) {
    b_in1->size[0] = in1->size[0];
  } else if (in5->size[0] == 1) {
    b_in1->size[0] = in3->size[0];
  } else {
    b_in1->size[0] = in5->size[0];
  }
  emxEnsureCapacity_real_T(b_in1, i, &ih_emlrtRTEI);
  stride_0_0 = (in1->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  stride_2_0 = (in5->size[0] != 1);
  if (in5->size[0] == 1) {
    b = in3->size[0];
  } else {
    b = in5->size[0];
  }
  if (b == 1) {
    b = in1->size[0];
  } else if (in5->size[0] == 1) {
    b = in3->size[0];
  } else {
    b = in5->size[0];
  }
  for (i = 0; i < b; i++) {
    b_in1->data[i] =
        in1->data[i * stride_0_0] +
        in2 * (in3->data[i * stride_1_0] - in4->data[i * stride_1_0]) /
            (in5->data[i * stride_2_0] * in5->data[i * stride_2_0]);
  }
  i = in1->size[0];
  in1->size[0] = b_in1->size[0];
  emxEnsureCapacity_real_T(in1, i, &jh_emlrtRTEI);
  for (i = 0; i < b_in1->size[0]; i++) {
    in1->data[i] = b_in1->data[i];
  }
  emxFree_real_T(&b_in1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

static real_T rt_remd_snf(real_T u0, real_T u1)
{
  real_T y;
  if (std::isnan(u0) || std::isnan(u1) || std::isinf(u0)) {
    y = rtNaN;
  } else if (std::isinf(u1)) {
    y = u0;
  } else if ((u1 != 0.0) && (u1 != std::trunc(u1))) {
    real_T q;
    q = std::abs(u0 / u1);
    if (!(std::abs(q - std::floor(q + 0.5)) > DBL_EPSILON * q)) {
      y = 0.0 * u0;
    } else {
      y = std::fmod(u0, u1);
    }
  } else {
    y = std::fmod(u0, u1);
  }
  return y;
}

//
//
namespace coder {
void firls(real_T varargin_1, const real_T varargin_2[4], emxArray_real_T *h)
{
  static const int8_T A[4]{1, 1, 0, 0};
  emxArray_real_T gpu_a;
  emxArray_real_T *G;
  emxArray_real_T *b;
  emxArray_real_T *b0;
  emxArray_real_T *b_dv;
  emxArray_real_T *b_dv1;
  emxArray_real_T *b_y;
  emxArray_real_T *cpu_a;
  emxArray_real_T *k;
  emxArray_real_T *m;
  emxArray_real_T *sinc1A;
  emxArray_real_T *sinc2A;
  emxArray_real_T *sinc3A;
  emxArray_real_T *sinc4A;
  emxArray_real_T *x;
  emxArray_real_T *y;
  real_T f1;
  real_T max_freq;
  real_T min_freq;
  int32_T i;
  boolean_T Nodd;
  boolean_T a_outdatedOnCpu;
  boolean_T a_outdatedOnGpu;
  gpuEmxReset_real_T(&gpu_a);
  a_outdatedOnCpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  max_freq = 0.0;
  min_freq = 0.0;
  for (i = 0; i < 3; i++) {
    f1 = varargin_2[i + 1];
    if (std::isnan(f1)) {
      Nodd = false;
    } else if (std::isnan(max_freq)) {
      Nodd = true;
    } else {
      Nodd = (max_freq < f1);
    }
    if (Nodd) {
      max_freq = f1;
    }
    if (std::isnan(f1)) {
      Nodd = false;
    } else if (std::isnan(min_freq)) {
      Nodd = true;
    } else {
      Nodd = (min_freq > f1);
    }
    if (Nodd) {
      min_freq = f1;
    }
  }
  emxInit_real_T(&m, 2, &be_emlrtRTEI, true);
  emxInit_real_T(&k, 1, &ce_emlrtRTEI, true);
  emxInit_real_T(&G, 2, &re_emlrtRTEI, true);
  emxInit_real_T(&sinc1A, 2, &se_emlrtRTEI, true);
  emxInit_real_T(&sinc2A, 2, &te_emlrtRTEI, true);
  emxInit_real_T(&sinc3A, 2, &ue_emlrtRTEI, true);
  emxInit_real_T(&sinc4A, 2, &ve_emlrtRTEI, true);
  emxInit_real_T(&b, 1, &ie_emlrtRTEI, true);
  emxInit_real_T(&cpu_a, 1, &we_emlrtRTEI, true);
  emxInit_real_T(&b_dv, 1, &xe_emlrtRTEI, true);
  emxInit_real_T(&x, 1, &ye_emlrtRTEI, true);
  emxInit_real_T(&b_dv1, 1, &xe_emlrtRTEI, true);
  emxInit_real_T(&y, 1, &af_emlrtRTEI, true);
  emxInit_real_T(&b_y, 1, &bf_emlrtRTEI, true);
  emxInit_real_T(&b0, 1, &ke_emlrtRTEI, true);
  if ((!(max_freq > 1.0)) && (!(min_freq < 0.0))) {
    real_T F[4];
    real_T L;
    real_T b_b0;
    real_T tmpStorageLen;
    int32_T b_i;
    int32_T i1Start;
    int32_T loop_ub;
    int32_T md2;
    int32_T nG;
    int32_T nx;
    F[0] = varargin_2[0] / 2.0;
    F[1] = varargin_2[1] / 2.0;
    F[2] = varargin_2[2] / 2.0;
    F[3] = varargin_2[3] / 2.0;
    L = ((varargin_1 + 1.0) - 1.0) / 2.0;
    Nodd = (rt_remd_snf(varargin_1 + 1.0, 2.0) == 1.0);
    b_b0 = 0.0;
    if (!Nodd) {
      b_i = m->size[0] * m->size[1];
      m->size[0] = 1;
      m->size[1] = static_cast<int32_T>(L) + 1;
      emxEnsureCapacity_real_T(m, b_i, &be_emlrtRTEI);
      for (b_i = 0; b_i <= static_cast<int32_T>(L); b_i++) {
        m->data[b_i] = static_cast<real_T>(b_i) + 0.5;
      }
    } else {
      b_i = m->size[0] * m->size[1];
      m->size[0] = 1;
      m->size[1] = static_cast<int32_T>(L) + 1;
      emxEnsureCapacity_real_T(m, b_i, &ae_emlrtRTEI);
      for (b_i = 0; b_i <= static_cast<int32_T>(L); b_i++) {
        m->data[b_i] = b_i;
      }
    }
    b_i = k->size[0];
    k->size[0] = m->size[1];
    emxEnsureCapacity_real_T(k, b_i, &ce_emlrtRTEI);
    for (b_i = 0; b_i < m->size[1]; b_i++) {
      k->data[b_i] = m->data[b_i];
    }
    a_outdatedOnGpu = (F[2] - F[1] != 0.0);
    nG = k->size[0];
    if (a_outdatedOnGpu) {
      b_i = G->size[0] * G->size[1];
      G->size[0] = k->size[0];
      G->size[1] = k->size[0];
      emxEnsureCapacity_real_T(G, b_i, &de_emlrtRTEI);
      for (b_i = 0; b_i < k->size[0] * k->size[0]; b_i++) {
        G->data[b_i] = 0.0;
      }
      tmpStorageLen = 2.0 * static_cast<real_T>(k->size[0]) - 1.0;
    } else {
      G->size[0] = 0;
      G->size[1] = 0;
      tmpStorageLen = 0.0;
    }
    b_i = sinc1A->size[0] * sinc1A->size[1];
    sinc1A->size[0] = 1;
    sinc1A->size[1] = static_cast<int32_T>(tmpStorageLen);
    emxEnsureCapacity_real_T(sinc1A, b_i, &ee_emlrtRTEI);
    b_i = sinc2A->size[0] * sinc2A->size[1];
    sinc2A->size[0] = 1;
    sinc2A->size[1] = static_cast<int32_T>(tmpStorageLen);
    emxEnsureCapacity_real_T(sinc2A, b_i, &fe_emlrtRTEI);
    b_i = sinc3A->size[0] * sinc3A->size[1];
    sinc3A->size[0] = 1;
    sinc3A->size[1] = static_cast<int32_T>(tmpStorageLen);
    emxEnsureCapacity_real_T(sinc3A, b_i, &ge_emlrtRTEI);
    b_i = sinc4A->size[0] * sinc4A->size[1];
    sinc4A->size[0] = 1;
    sinc4A->size[1] = static_cast<int32_T>(tmpStorageLen);
    emxEnsureCapacity_real_T(sinc4A, b_i, &he_emlrtRTEI);
    if (Nodd) {
      i1Start = -1;
      if (k->size[0] < 2) {
        i = 0;
        nx = 0;
      } else {
        i = 1;
        nx = k->size[0];
      }
      loop_ub = nx - i;
      for (b_i = 0; b_i < loop_ub; b_i++) {
        k->data[b_i] = k->data[i + b_i];
      }
      b_i = k->size[0];
      k->size[0] = nx - i;
      emxEnsureCapacity_real_T(k, b_i, &je_emlrtRTEI);
    } else {
      i1Start = 0;
    }
    b_i = b->size[0];
    b->size[0] = k->size[0];
    emxEnsureCapacity_real_T(b, b_i, &ie_emlrtRTEI);
    for (b_i = 0; b_i < k->size[0]; b_i++) {
      b->data[b_i] = 0.0;
    }
    for (int32_T s{0}; s < 2; s++) {
      real_T a;
      real_T b1;
      int8_T i1;
      i = s << 1;
      i1 = A[i];
      f1 = F[i + 1];
      max_freq = F[i];
      min_freq = static_cast<real_T>(A[i + 1] - i1) / (f1 - max_freq);
      b1 = static_cast<real_T>(i1) - min_freq * max_freq;
      if (Nodd) {
        b_b0 += b1 * (f1 - max_freq) +
                min_freq / 2.0 * (f1 * f1 - max_freq * max_freq);
      }
      b_i = b_dv->size[0];
      b_dv->size[0] = k->size[0];
      emxEnsureCapacity_real_T(b_dv, b_i, &me_emlrtRTEI);
      for (b_i = 0; b_i < k->size[0]; b_i++) {
        b_dv->data[b_i] = 6.2831853071795862 * k->data[b_i] * F[i + 1];
      }
      nx = b_dv->size[0];
      b_i = b_dv1->size[0];
      b_dv1->size[0] = k->size[0];
      emxEnsureCapacity_real_T(b_dv1, b_i, &me_emlrtRTEI);
      for (loop_ub = 0; loop_ub < nx; loop_ub++) {
        b_dv->data[loop_ub] = std::cos(b_dv->data[loop_ub]);
        b_dv1->data[loop_ub] = 6.2831853071795862 * k->data[loop_ub] * F[i];
      }
      nx = b_dv1->size[0];
      for (loop_ub = 0; loop_ub < nx; loop_ub++) {
        b_dv1->data[loop_ub] = std::cos(b_dv1->data[loop_ub]);
      }
      a = min_freq / 39.478417604357432;
      if (b_dv->size[0] == 1) {
        b_i = k->size[0];
      } else {
        b_i = b_dv->size[0];
      }
      if ((b_dv->size[0] == k->size[0]) && (b->size[0] == b_i)) {
        for (b_i = 0; b_i < b->size[0]; b_i++) {
          b->data[b_i] += a * (b_dv->data[b_i] - b_dv1->data[b_i]) /
                          (k->data[b_i] * k->data[b_i]);
        }
      } else {
        binary_expand_op_7(b, a, b_dv, b_dv1, k);
      }
      b_i = y->size[0];
      y->size[0] = k->size[0];
      emxEnsureCapacity_real_T(y, b_i, &me_emlrtRTEI);
      for (b_i = 0; b_i < k->size[0]; b_i++) {
        y->data[b_i] = 2.0 * k->data[b_i] * F[i + 1];
      }
      f1 = F[i + 1];
      a = f1 * (min_freq * f1 + b1);
      md2 = y->size[0];
      b_i = b_y->size[0];
      b_y->size[0] = k->size[0];
      emxEnsureCapacity_real_T(b_y, b_i, &me_emlrtRTEI);
      for (loop_ub = 0; loop_ub < md2; loop_ub++) {
        if (std::abs(y->data[loop_ub]) < 1.0020841800044864E-292) {
          y->data[loop_ub] = 1.0;
        } else {
          f1 = 3.1415926535897931 * y->data[loop_ub];
          f1 = std::sin(f1) / f1;
          y->data[loop_ub] = f1;
        }
        b_y->data[loop_ub] = 2.0 * k->data[loop_ub] * F[i];
      }
      f1 = F[i];
      max_freq = f1 * (min_freq * f1 + b1);
      nx = b_y->size[0];
      for (loop_ub = 0; loop_ub < nx; loop_ub++) {
        if (std::abs(b_y->data[loop_ub]) < 1.0020841800044864E-292) {
          b_y->data[loop_ub] = 1.0;
        } else {
          f1 = 3.1415926535897931 * b_y->data[loop_ub];
          f1 = std::sin(f1) / f1;
          b_y->data[loop_ub] = f1;
        }
      }
      if (b->size[0] == y->size[0]) {
        for (b_i = 0; b_i < b->size[0]; b_i++) {
          b->data[b_i] += a * y->data[b_i] - max_freq * b_y->data[b_i];
        }
      } else {
        binary_expand_op_6(b, a, y, max_freq, b_y);
      }
      if (a_outdatedOnGpu) {
        real_T f2;
        uint32_T b_i1;
        f1 = 2.0 * F[i + 1];
        f2 = 2.0 * F[i];
        nx = static_cast<int32_T>(tmpStorageLen);
        for (md2 = 0; md2 < nx; md2++) {
          b_i1 = (static_cast<uint32_T>(md2) + static_cast<uint32_T>(i1Start)) +
                 1U;
          max_freq = (static_cast<real_T>(md2) + 1.0) - static_cast<real_T>(nG);
          min_freq = f1 * static_cast<real_T>(b_i1);
          b1 = f2 * static_cast<real_T>(b_i1);
          a = f1 * max_freq;
          max_freq *= f2;
          if (std::abs(min_freq) < 1.0020841800044864E-292) {
            sinc1A->data[md2] = 1.0;
          } else {
            min_freq *= 3.1415926535897931;
            sinc1A->data[md2] = std::sin(min_freq) / min_freq;
          }
          if (std::abs(b1) < 1.0020841800044864E-292) {
            sinc2A->data[md2] = 1.0;
          } else {
            b1 *= 3.1415926535897931;
            sinc2A->data[md2] = std::sin(b1) / b1;
          }
          if (std::abs(a) < 1.0020841800044864E-292) {
            sinc3A->data[md2] = 1.0;
          } else {
            a *= 3.1415926535897931;
            sinc3A->data[md2] = std::sin(a) / a;
          }
          if (std::abs(max_freq) < 1.0020841800044864E-292) {
            sinc4A->data[md2] = 1.0;
          } else {
            max_freq *= 3.1415926535897931;
            sinc4A->data[md2] = std::sin(max_freq) / max_freq;
          }
        }
        for (md2 = 0; md2 < nG; md2++) {
          for (nx = 0; nx < nG; nx++) {
            b_i1 =
                (static_cast<uint32_T>(nx) + static_cast<uint32_T>(md2)) + 1U;
            max_freq = static_cast<real_T>(nx - md2) + static_cast<real_T>(nG);
            G->data[nx + G->size[0] * md2] +=
                0.25 *
                (f1 * (sinc1A->data[static_cast<int32_T>(b_i1) - 1] +
                       sinc3A->data[static_cast<int32_T>(max_freq) - 1]) -
                 f2 * (sinc2A->data[static_cast<int32_T>(b_i1) - 1] +
                       sinc4A->data[static_cast<int32_T>(max_freq) - 1]));
          }
        }
      }
    }
    if (Nodd) {
      b_i = b0->size[0];
      b0->size[0] = b->size[0] + 1;
      emxEnsureCapacity_real_T(b0, b_i, &ke_emlrtRTEI);
      b0->data[0] = b_b0;
      loop_ub = b->size[0];
      for (b_i = 0; b_i < loop_ub; b_i++) {
        b0->data[b_i + 1] = b->data[b_i];
      }
      b_i = b->size[0];
      b->size[0] = b0->size[0];
      emxEnsureCapacity_real_T(b, b_i, &oe_emlrtRTEI);
      for (b_i = 0; b_i < b0->size[0]; b_i++) {
        b->data[b_i] = b0->data[b_i];
      }
    }
    if (a_outdatedOnGpu) {
      mldivide(G, b, cpu_a, &a_outdatedOnCpu, &gpu_a, &a_outdatedOnGpu);
    } else {
      b_i = cpu_a->size[0];
      cpu_a->size[0] = b->size[0];
      emxEnsureCapacity_real_T(cpu_a, b_i, &le_emlrtRTEI);
      for (b_i = 0; b_i < b->size[0]; b_i++) {
        cpu_a->data[b_i] = 4.0 * b->data[b_i];
      }
      if (Nodd) {
        cpu_a->data[0] /= 2.0;
      }
    }
    if (Nodd) {
      if (L + 1.0 < 2.0) {
        md2 = 0;
        nx = 1;
        i = -1;
        nG = 0;
        i1Start = 0;
      } else {
        md2 = static_cast<int32_T>(L + 1.0) - 1;
        nx = -1;
        i = 1;
        nG = 1;
        i1Start = static_cast<int32_T>(L + 1.0);
      }
      b_i = h->size[0] * h->size[1];
      h->size[0] = 1;
      h->size[1] = ((div_s32(i - md2, nx) + i1Start) - nG) + 2;
      emxEnsureCapacity_real_T(h, b_i, &pe_emlrtRTEI);
      loop_ub = div_s32(i - md2, nx);
      for (b_i = 0; b_i <= loop_ub; b_i++) {
        if (a_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
        }
        a_outdatedOnCpu = false;
        h->data[b_i] = cpu_a->data[md2 + nx * b_i] / 2.0;
      }
      if (a_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
      }
      h->data[div_s32(i - md2, nx) + 1] = cpu_a->data[0];
      loop_ub = i1Start - nG;
      for (b_i = 0; b_i < loop_ub; b_i++) {
        h->data[(b_i + div_s32(i - md2, nx)) + 2] = cpu_a->data[nG + b_i] / 2.0;
      }
    } else {
      b_i = x->size[0];
      x->size[0] = cpu_a->size[0];
      emxEnsureCapacity_real_T(x, b_i, &ne_emlrtRTEI);
      for (b_i = 0; b_i < cpu_a->size[0]; b_i++) {
        if (a_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
        }
        a_outdatedOnCpu = false;
        x->data[b_i] = cpu_a->data[b_i];
      }
      nx = cpu_a->size[0] - 1;
      md2 = cpu_a->size[0] >> 1;
      for (i = 0; i < md2; i++) {
        max_freq = x->data[i];
        x->data[i] = x->data[nx - i];
        x->data[nx - i] = max_freq;
      }
      b_i = h->size[0] * h->size[1];
      h->size[0] = 1;
      h->size[1] = x->size[0] + cpu_a->size[0];
      emxEnsureCapacity_real_T(h, b_i, &qe_emlrtRTEI);
      loop_ub = x->size[0];
      for (b_i = 0; b_i < loop_ub; b_i++) {
        h->data[b_i] = 0.5 * x->data[b_i];
      }
      loop_ub = cpu_a->size[0];
      for (b_i = 0; b_i < loop_ub; b_i++) {
        if (a_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
        }
        a_outdatedOnCpu = false;
        h->data[b_i + x->size[0]] = 0.5 * cpu_a->data[b_i];
      }
    }
  }
  emxFree_real_T(&b0);
  emxFree_real_T(&b_y);
  emxFree_real_T(&y);
  emxFree_real_T(&b_dv1);
  emxFree_real_T(&x);
  emxFree_real_T(&b_dv);
  emxFree_real_T(&cpu_a);
  emxFree_real_T(&b);
  emxFree_real_T(&sinc4A);
  emxFree_real_T(&sinc3A);
  emxFree_real_T(&sinc2A);
  emxFree_real_T(&sinc1A);
  emxFree_real_T(&G);
  emxFree_real_T(&k);
  emxFree_real_T(&m);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_a);
}

} // namespace coder

// End of code generation (firls.cu)

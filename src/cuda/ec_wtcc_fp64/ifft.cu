//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ifft.cu
//
// Code generation for function 'ifft'
//

// Include files
#include "ifft.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "hipfft/hipfft.h"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo vd_emlrtRTEI{
    83,                                                              // lineNo
    1,                                                               // colNo
    "ifft",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/ifft.m" // pName
};

// Function Definitions
//
//
namespace coder {
void ifft(const emxArray_creal_T *x, emxArray_creal_T *y)
{
  emxArray_creal_T gpu_x;
  emxArray_creal_T *cpu_x;
  gpuEmxReset_creal_T(&gpu_x);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((x->size[0] == 0) || (x->size[1] == 0)) {
    int32_T i;
    uint32_T ySize_idx_0;
    uint32_T ySize_idx_1;
    ySize_idx_0 = static_cast<uint32_T>(x->size[0]);
    ySize_idx_1 = static_cast<uint32_T>(x->size[1]);
    i = y->size[0] * y->size[1];
    y->size[0] = x->size[0];
    y->size[1] = x->size[1];
    emxEnsureCapacity_creal_T(y, i, &vd_emlrtRTEI);
    if (static_cast<int32_T>(ySize_idx_0) * static_cast<int32_T>(ySize_idx_1) -
            1 >=
        0) {
      std::memset(&y->data[0], 0,
                  static_cast<uint32_T>(static_cast<int32_T>(ySize_idx_0) *
                                        static_cast<int32_T>(ySize_idx_1)) *
                      sizeof(creal_T));
    }
  } else {
    hipfftHandle fftPlanHandle;
    int32_T i;
    int32_T inembed;
    boolean_T x_outdatedOnGpu;
    emxInit_creal_T(&cpu_x, 2, &ud_emlrtRTEI, true);
    x_outdatedOnGpu = false;
    i = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = x->size[0];
    cpu_x->size[1] = x->size[1];
    emxEnsureCapacity_creal_T(cpu_x, i, &ud_emlrtRTEI);
    for (i = 0; i < x->size[0] * x->size[1]; i++) {
      cpu_x->data[i] = x->data[i];
      x_outdatedOnGpu = true;
    }
    inembed = x->size[1];
    fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, x->size[0], 1,
                                     HIPFFT_Z2Z, x->size[0]);
    gpuEmxEnsureCapacity_creal_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    if (x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_x, cpu_x);
    }
    hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&gpu_x.data[0],
                 (hipfftDoubleComplex *)&gpu_x.data[0], HIPFFT_BACKWARD);
    x_outdatedOnGpu = true;
    i = y->size[0] * y->size[1];
    y->size[0] = cpu_x->size[0];
    y->size[1] = cpu_x->size[1];
    emxEnsureCapacity_creal_T(y, i, &vd_emlrtRTEI);
    inembed = x->size[1];
    for (i = 0; i < cpu_x->size[0] * cpu_x->size[1]; i++) {
      real_T ai;
      real_T ar;
      if (x_outdatedOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal_T(cpu_x, &gpu_x);
      }
      ar = cpu_x->data[i].re;
      x_outdatedOnGpu = false;
      ai = cpu_x->data[i].im;
      if (ai == 0.0) {
        y->data[i].re = ar / static_cast<real_T>(inembed);
        y->data[i].im = 0.0;
      } else if (ar == 0.0) {
        y->data[i].re = 0.0;
        y->data[i].im = ai / static_cast<real_T>(inembed);
      } else {
        y->data[i].re = ar / static_cast<real_T>(inembed);
        y->data[i].im = ai / static_cast<real_T>(inembed);
      }
    }
    emxFree_creal_T(&cpu_x);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_creal_T(&gpu_x);
}

} // namespace coder

// End of code generation (ifft.cu)

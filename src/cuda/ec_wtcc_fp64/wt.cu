#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wt.cu
//
// Code generation for function 'wt'
//

// Include files
#include "wt.h"
#include "bsxfun.h"
#include "colon.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "fft.h"
#include "ifft.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo pd_emlrtRTEI{
    188,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo qd_emlrtRTEI{
    205,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo rd_emlrtRTEI{
    209,  // lineNo
    9,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo sd_emlrtRTEI{
    125,                                                          // lineNo
    9,                                                            // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

// Function Declarations
static __global__ void
cwtfilterbank_wt_kernel26(const int32_T i, const int32_T b, emxArray_real_T x);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel26(
    const int32_T i, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T xtmp;
    int32_T b_j1;
    int32_T j2;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

//
//
namespace coder {
void cwtfilterbank_wt(const cwtfilterbank *self, const emxArray_real_T *x,
                      emxArray_creal_T *varargout_1,
                      emxArray_real_T *varargout_2,
                      emxArray_real_T *varargout_3)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T gpu_xposdft;
  emxArray_creal_T *cpu_xposdft;
  emxArray_creal_T *r;
  emxArray_real_T gpu_r1;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xv;
  emxArray_real_T *cpu_r1;
  emxArray_real_T *cpu_x;
  emxArray_real_T *cpu_xv;
  emxArray_real_T *samples;
  real_T M;
  real_T Norig;
  real_T cf;
  real_T d3;
  real_T sigmaPsi;
  int32_T b_i;
  int32_T i;
  int32_T i1;
  int32_T i3;
  int32_T i5;
  int32_T nd2;
  char_T wname_idx_0;
  boolean_T r1_outdatedOnCpu;
  boolean_T r1_outdatedOnGpu;
  boolean_T xv_outdatedOnCpu;
  boolean_T xv_outdatedOnGpu;
  gpuEmxReset_real_T(&gpu_r1);
  gpuEmxReset_real_T(&gpu_x);
  gpuEmxReset_creal_T(&gpu_xposdft);
  gpuEmxReset_real_T(&gpu_xv);
  r1_outdatedOnCpu = false;
  r1_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&cpu_xv, 2, &hb_emlrtRTEI, true);
  xv_outdatedOnCpu = false;
  xv_outdatedOnGpu = false;
  i = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = x->size[0];
  emxEnsureCapacity_real_T(cpu_xv, i, &hb_emlrtRTEI);
  for (i = 0; i < x->size[0]; i++) {
    cpu_xv->data[i] = x->data[i];
    xv_outdatedOnGpu = true;
  }
  Norig = self->SignalLength;
  if (self->SignalPad > 0.0) {
    boolean_T validLaunchParams;
    M = self->SignalPad;
    if (M < 1.0) {
      b_i = 0;
    } else {
      b_i = static_cast<int32_T>(M);
    }
    M = (static_cast<real_T>(x->size[0]) - self->SignalPad) + 1.0;
    if (M > x->size[0]) {
      i1 = 0;
      i3 = 1;
      i5 = -1;
    } else {
      i1 = x->size[0] - 1;
      i3 = -1;
      i5 = static_cast<int32_T>(M) - 1;
    }
    emxInit_real_T(&cpu_x, 2, &ib_emlrtRTEI, true);
    r1_outdatedOnCpu = false;
    i = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = 1;
    cpu_x->size[1] = b_i;
    emxEnsureCapacity_real_T(cpu_x, i, &ib_emlrtRTEI);
    for (i = 0; i < b_i; i++) {
      cpu_x->data[i] = x->data[i];
      r1_outdatedOnCpu = true;
    }
    mwGetLaunchParameters1D(computeNumIters((b_i >> 1) - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !r1_outdatedOnCpu);
    if (r1_outdatedOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      cwtfilterbank_wt_kernel26<<<grid, block>>>(b_i, (b_i >> 1) - 1, gpu_x);
    }
    r1_outdatedOnCpu = true;
    xv_outdatedOnGpu = false;
    i = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    cpu_xv->size[1] =
        ((x->size[0] + cpu_x->size[1]) + div_s32(i5 - i1, i3)) + 1;
    emxEnsureCapacity_real_T(cpu_xv, i, &lb_emlrtRTEI);
    b_i = cpu_x->size[1];
    for (i = 0; i < b_i; i++) {
      if (r1_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_x, &gpu_x);
      }
      r1_outdatedOnCpu = false;
      cpu_xv->data[i] = cpu_x->data[i];
      xv_outdatedOnGpu = true;
    }
    b_i = x->size[0];
    for (i = 0; i < b_i; i++) {
      cpu_xv->data[i + cpu_x->size[1]] = x->data[i];
      xv_outdatedOnGpu = true;
    }
    b_i = div_s32(i5 - i1, i3);
    for (i = 0; i <= b_i; i++) {
      cpu_xv->data[(i + x->size[0]) + cpu_x->size[1]] = x->data[i1 + i3 * i];
      xv_outdatedOnGpu = true;
    }
    emxFree_real_T(&cpu_x);
  }
  emxInit_creal_T(&cpu_xposdft, 2, &qb_emlrtRTEI, true);
  fft(cpu_xv, &xv_outdatedOnCpu, &gpu_xv, &xv_outdatedOnGpu, cpu_xposdft,
      &r1_outdatedOnCpu, &gpu_xposdft, &r1_outdatedOnGpu);
  emxFree_real_T(&cpu_xv);
  emxInit_creal_T(&r, 2, &vb_emlrtRTEI, true);
  if (r1_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_creal_T(cpu_xposdft, &gpu_xposdft);
  }
  bsxfun(cpu_xposdft, self->PsiDFT, r);
  emxFree_creal_T(&cpu_xposdft);
  ifft(r, varargout_1);
  emxFree_creal_T(&r);
  if (self->SignalPad > 0.0) {
    M = self->SignalPad + 1.0;
    d3 = self->SignalPad + Norig;
    if (M > d3) {
      nd2 = 0;
      i1 = 0;
    } else {
      nd2 = static_cast<int32_T>(M) - 1;
      i1 = static_cast<int32_T>(d3);
    }
    i3 = varargout_1->size[0];
    b_i = i1 - nd2;
    for (i = 0; i < b_i; i++) {
      for (i5 = 0; i5 < i3; i5++) {
        varargout_1->data[i5 + i3 * i] =
            varargout_1->data[i5 + varargout_1->size[0] * (nd2 + i)];
      }
    }
    i = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[1] = i1 - nd2;
    emxEnsureCapacity_creal_T(varargout_1, i, &jb_emlrtRTEI);
  }
  i = varargout_2->size[0];
  varargout_2->size[0] = self->WaveletCenterFrequencies->size[0];
  emxEnsureCapacity_real_T(varargout_2, i, &pd_emlrtRTEI);
  for (i = 0; i < self->WaveletCenterFrequencies->size[0]; i++) {
    varargout_2->data[i] = self->WaveletCenterFrequencies->data[i];
  }
  wname_idx_0 = cv[static_cast<uint8_T>(self->Wavelet[0]) & 127];
  if (wname_idx_0 == 'm') {
    cf = rtNaN;
  } else if (wname_idx_0 == 'a') {
    cf = 6.0;
    sigmaPsi = 1.4142135623730951;
  } else {
    cf = 5.0;
    sigmaPsi = 5.847705;
  }
  emxInit_real_T(&samples, 1, &qd_emlrtRTEI, true);
  i = samples->size[0];
  samples->size[0] = static_cast<int32_T>(Norig);
  emxEnsureCapacity_real_T(samples, i, &qd_emlrtRTEI);
  for (i = 0; i < static_cast<int32_T>(Norig); i++) {
    samples->data[i] = 0.0;
  }
  if (std::isinf(Norig)) {
    M = rtNaN;
  } else if (Norig == 0.0) {
    M = 0.0;
  } else {
    M = std::fmod(Norig, 2.0);
  }
  if (M == 1.0) {
    M = std::ceil(Norig / 2.0);
    if (!(M < 1.0)) {
      for (i = 0; i <= static_cast<int32_T>(M - 1.0); i++) {
        samples->data[i] = static_cast<real_T>(i) + 1.0;
      }
    }
    if (M + 1.0 > Norig) {
      nd2 = 0;
    } else {
      nd2 = static_cast<int32_T>(static_cast<uint32_T>(M));
    }
    if (!(M - 1.0 < 1.0)) {
      for (i = 0; i <= static_cast<int32_T>(-(1.0 - (M - 1.0))); i++) {
        samples->data[nd2 + i] = (M - 1.0) - static_cast<real_T>(i);
      }
    }
  } else {
    M = Norig / 2.0;
    if (!(M < 1.0)) {
      for (i = 0; i <= static_cast<int32_T>(M - 1.0); i++) {
        samples->data[i] = static_cast<real_T>(i) + 1.0;
      }
    }
    M = Norig / 2.0 + 1.0;
    if (M > Norig) {
      nd2 = 0;
    } else {
      nd2 = static_cast<int32_T>(M) - 1;
    }
    M = Norig / 2.0;
    if (!(M < 1.0)) {
      if (std::floor(M) == M) {
        for (i = 0; i <= static_cast<int32_T>(-(1.0 - M)); i++) {
          samples->data[nd2 + i] = M - static_cast<real_T>(i);
        }
      } else {
        emxInit_real_T(&cpu_r1, 2, &sd_emlrtRTEI, true);
        eml_float_colon(M, cpu_r1, &r1_outdatedOnCpu, &gpu_r1,
                        &r1_outdatedOnGpu);
        for (i = 0; i < cpu_r1->size[1]; i++) {
          if (r1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_r1, &gpu_r1);
          }
          r1_outdatedOnCpu = false;
          samples->data[nd2 + i] = cpu_r1->data[i];
        }
        emxFree_real_T(&cpu_r1);
      }
    }
  }
  M = 6.2831853071795862 / cf / sigmaPsi * (1.0 / self->SamplingFrequency);
  i = varargout_3->size[0];
  varargout_3->size[0] = samples->size[0];
  emxEnsureCapacity_real_T(varargout_3, i, &rd_emlrtRTEI);
  for (i = 0; i < samples->size[0]; i++) {
    varargout_3->data[i] = 1.0 / (M * samples->data[i]);
  }
  emxFree_real_T(&samples);
  nd2 = self->WaveletCenterFrequencies->size[0];
  M = self->WaveletCenterFrequencies->data[0];
  d3 = self->WaveletCenterFrequencies->data[0];
  for (b_i = 0; b_i <= nd2 - 2; b_i++) {
    cf = self->WaveletCenterFrequencies->data[b_i + 1];
    if (std::isnan(cf)) {
      r1_outdatedOnCpu = false;
    } else if (std::isnan(M)) {
      r1_outdatedOnCpu = true;
    } else {
      r1_outdatedOnCpu = (M < cf);
    }
    if (r1_outdatedOnCpu) {
      M = cf;
    }
    if (std::isnan(cf)) {
      r1_outdatedOnCpu = false;
    } else if (std::isnan(d3)) {
      r1_outdatedOnCpu = true;
    } else {
      r1_outdatedOnCpu = (d3 < cf);
    }
    if (r1_outdatedOnCpu) {
      d3 = cf;
    }
  }
  nd2 = varargout_3->size[0] - 1;
  for (b_i = 0; b_i <= nd2; b_i++) {
    if (varargout_3->data[b_i] > M) {
      varargout_3->data[b_i] = d3;
    }
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_xv);
  gpuEmxFree_creal_T(&gpu_xposdft);
  gpuEmxFree_real_T(&gpu_x);
  gpuEmxFree_real_T(&gpu_r1);
}

} // namespace coder

// End of code generation (wt.cu)

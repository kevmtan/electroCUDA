#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// casyi.cu
//
// Code generation for function 'casyi'
//

// Include files
#include "casyi.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_mexutil.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
int32_T casyi(const creal_T z, real_T fnu, int32_T kode, creal_T *y)
{
  real_T absxi;
  real_T absxr;
  real_T ak1_im;
  real_T ak1_re;
  real_T cz_im;
  real_T yr;
  int32_T nz;
  nz = 0;
  if (z.im == 0.0) {
    ak1_re = 0.15915494309189535 / z.re;
    ak1_im = 0.0;
  } else if (z.re == 0.0) {
    ak1_re = 0.0;
    ak1_im = -(0.15915494309189535 / z.im);
  } else {
    ak1_im = std::abs(z.re);
    absxi = std::abs(z.im);
    if (ak1_im > absxi) {
      absxi = z.im / z.re;
      absxr = z.re + absxi * z.im;
      ak1_re = (absxi * 0.0 + 0.15915494309189535) / absxr;
      ak1_im = (0.0 - absxi * 0.15915494309189535) / absxr;
    } else if (absxi == ak1_im) {
      if (z.re > 0.0) {
        absxi = 0.5;
      } else {
        absxi = -0.5;
      }
      if (z.im > 0.0) {
        absxr = 0.5;
      } else {
        absxr = -0.5;
      }
      ak1_re = (0.15915494309189535 * absxi + 0.0 * absxr) / ak1_im;
      ak1_im = (0.0 * absxi - 0.15915494309189535 * absxr) / ak1_im;
    } else {
      absxi = z.re / z.im;
      absxr = z.im + absxi * z.re;
      ak1_re = absxi * 0.15915494309189535 / absxr;
      ak1_im = (absxi * 0.0 - 0.15915494309189535) / absxr;
    }
  }
  if (ak1_im == 0.0) {
    if (ak1_re < 0.0) {
      yr = 0.0;
      absxi = std::sqrt(-ak1_re);
    } else {
      yr = std::sqrt(ak1_re);
      absxi = 0.0;
    }
  } else if (ak1_re == 0.0) {
    if (ak1_im < 0.0) {
      yr = std::sqrt(-ak1_im / 2.0);
      absxi = -yr;
    } else {
      yr = std::sqrt(ak1_im / 2.0);
      absxi = yr;
    }
  } else if (std::isnan(ak1_re)) {
    yr = rtNaN;
    absxi = rtNaN;
  } else if (std::isnan(ak1_im)) {
    yr = rtNaN;
    absxi = rtNaN;
  } else if (std::isinf(ak1_im)) {
    yr = std::abs(ak1_im);
    absxi = ak1_im;
  } else if (std::isinf(ak1_re)) {
    if (ak1_re < 0.0) {
      yr = 0.0;
      absxi = ak1_im * -ak1_re;
    } else {
      yr = ak1_re;
      absxi = 0.0;
    }
  } else {
    absxr = std::abs(ak1_re);
    absxi = std::abs(ak1_im);
    if ((absxr > 4.4942328371557893E+307) ||
        (absxi > 4.4942328371557893E+307)) {
      absxr *= 0.5;
      absxi = rt_hypotd_snf(absxr, absxi * 0.5);
      if (absxi > absxr) {
        yr = std::sqrt(absxi) * std::sqrt(absxr / absxi + 1.0);
      } else {
        yr = std::sqrt(absxi) * 1.4142135623730951;
      }
    } else {
      yr = std::sqrt((rt_hypotd_snf(absxr, absxi) + absxr) * 0.5);
    }
    if (ak1_re > 0.0) {
      absxi = 0.5 * (ak1_im / yr);
    } else {
      if (ak1_im < 0.0) {
        absxi = -yr;
      } else {
        absxi = yr;
      }
      yr = 0.5 * (ak1_im / absxi);
    }
  }
  if (kode == 2) {
    ak1_re = 0.0;
    cz_im = z.im;
    absxr = 0.0;
  } else {
    ak1_re = z.re;
    cz_im = z.im;
    absxr = z.re;
  }
  if (std::abs(absxr) > 700.92179369444591) {
    nz = -1;
    y->re = rtNaN;
    y->im = 0.0;
  } else {
    real_T aa;
    real_T aez;
    real_T ak;
    real_T b_re;
    real_T bb;
    real_T bk;
    real_T ck_im;
    real_T cs1_im;
    real_T cs1_re;
    real_T cs2_im;
    real_T cs2_re;
    real_T dk_im;
    real_T dk_re;
    real_T ez_im;
    real_T ez_re;
    real_T im;
    real_T p1_re;
    real_T re;
    real_T sqk;
    int32_T i;
    boolean_T exitg1;
    boolean_T negateSinpi;
    absxr = fnu + fnu;
    if (ak1_re == 0.0) {
      ak1_re = std::cos(cz_im);
      cz_im = std::sin(cz_im);
    } else if (cz_im == 0.0) {
      ak1_re = std::exp(ak1_re);
      cz_im = 0.0;
    } else if (std::isinf(cz_im) && std::isinf(ak1_re) && (ak1_re < 0.0)) {
      ak1_re = 0.0;
      cz_im = 0.0;
    } else {
      ak1_im = std::exp(ak1_re / 2.0);
      ak1_re = ak1_im * (ak1_im * std::cos(cz_im));
      cz_im = ak1_im * (ak1_im * std::sin(cz_im));
    }
    re = yr * ak1_re - absxi * cz_im;
    im = yr * cz_im + absxi * ak1_re;
    ak1_im = 0.0;
    if (absxr > 4.7170688552396617E-153) {
      ak1_im = absxr * absxr;
    }
    ez_re = 8.0 * z.re;
    ez_im = 8.0 * z.im;
    aez = 8.0 * rt_hypotd_snf(z.re, z.im);
    if (z.im != 0.0) {
      absxr = fnu - static_cast<real_T>(static_cast<int32_T>(fnu));
      if (absxr < 0.0) {
        absxr = -absxr;
        negateSinpi = true;
      } else {
        negateSinpi = false;
      }
      if (absxr < 0.25) {
        bk = std::cos(absxr * 3.1415926535897931);
        p1_re = std::sin(absxr * 3.1415926535897931);
      } else if (absxr < 0.75) {
        absxi = (0.5 - absxr) * 3.1415926535897931;
        bk = std::sin(absxi);
        p1_re = std::cos(absxi);
      } else if (absxr < 1.25) {
        absxi = (1.0 - absxr) * 3.1415926535897931;
        bk = -std::cos(absxi);
        p1_re = std::sin(absxi);
      } else {
        absxi = (absxr - 1.5) * 3.1415926535897931;
        bk = std::sin(absxi);
        p1_re = -std::cos(absxi);
      }
      if (negateSinpi) {
        p1_re = -p1_re;
      }
      if (z.im < 0.0) {
        bk = -bk;
      }
      if (static_cast<int32_T>(fnu) != 0) {
        bk = -bk;
      } else {
        p1_re = -p1_re;
      }
    } else {
      p1_re = 0.0;
      bk = 0.0;
    }
    sqk = ak1_im - 1.0;
    yr = 2.2204460492503131E-16 / aez * std::abs(ak1_im - 1.0);
    ak1_re = 1.0;
    cs1_re = 1.0;
    cs1_im = 0.0;
    cs2_re = 1.0;
    cs2_im = 0.0;
    cz_im = 1.0;
    ck_im = 0.0;
    ak = 0.0;
    aa = 1.0;
    bb = aez;
    dk_re = ez_re;
    dk_im = ez_im;
    negateSinpi = true;
    i = 1;
    exitg1 = false;
    while ((!exitg1) && (i < 46)) {
      cz_im *= sqk;
      ck_im *= sqk;
      if (dk_im == 0.0) {
        if (ck_im == 0.0) {
          b_re = cz_im / dk_re;
          ck_im = 0.0;
        } else if (cz_im == 0.0) {
          b_re = 0.0;
          ck_im /= dk_re;
        } else {
          b_re = cz_im / dk_re;
          ck_im /= dk_re;
        }
      } else if (dk_re == 0.0) {
        if (cz_im == 0.0) {
          b_re = ck_im / dk_im;
          ck_im = 0.0;
        } else if (ck_im == 0.0) {
          b_re = 0.0;
          ck_im = -(cz_im / dk_im);
        } else {
          b_re = ck_im / dk_im;
          ck_im = -(cz_im / dk_im);
        }
      } else {
        ak1_im = std::abs(dk_re);
        absxi = std::abs(dk_im);
        if (ak1_im > absxi) {
          absxi = dk_im / dk_re;
          absxr = dk_re + absxi * dk_im;
          b_re = (cz_im + absxi * ck_im) / absxr;
          ck_im = (ck_im - absxi * cz_im) / absxr;
        } else if (absxi == ak1_im) {
          if (dk_re > 0.0) {
            absxi = 0.5;
          } else {
            absxi = -0.5;
          }
          if (dk_im > 0.0) {
            absxr = 0.5;
          } else {
            absxr = -0.5;
          }
          b_re = (cz_im * absxi + ck_im * absxr) / ak1_im;
          ck_im = (ck_im * absxi - cz_im * absxr) / ak1_im;
        } else {
          absxi = dk_re / dk_im;
          absxr = dk_im + absxi * dk_re;
          b_re = (absxi * cz_im + ck_im) / absxr;
          ck_im = (absxi * ck_im - cz_im) / absxr;
        }
      }
      cz_im = b_re;
      cs2_re += b_re;
      cs2_im += ck_im;
      ak1_re = -ak1_re;
      cs1_re += b_re * ak1_re;
      cs1_im += ck_im * ak1_re;
      dk_re += ez_re;
      dk_im += ez_im;
      aa = aa * std::abs(sqk) / bb;
      bb += aez;
      ak += 8.0;
      sqk -= ak;
      if (aa <= yr) {
        negateSinpi = false;
        exitg1 = true;
      } else {
        i++;
      }
    }
    if (negateSinpi) {
      nz = -2;
    } else {
      if (z.re + z.re < 700.92179369444591) {
        ak1_re = -2.0 * z.re;
        cz_im = -2.0 * z.im;
        if (ak1_re == 0.0) {
          ak1_re = std::cos(cz_im);
          cz_im = std::sin(cz_im);
        } else if (cz_im == 0.0) {
          ak1_re = std::exp(ak1_re);
          cz_im = 0.0;
        } else if (std::isinf(cz_im) && std::isinf(ak1_re) && (ak1_re < 0.0)) {
          ak1_re = 0.0;
          cz_im = 0.0;
        } else {
          ak1_im = std::exp(ak1_re / 2.0);
          ak1_re = ak1_im * (ak1_im * std::cos(cz_im));
          cz_im = ak1_im * (ak1_im * std::sin(cz_im));
        }
        b_re = ak1_re * cs2_re - cz_im * cs2_im;
        absxi = ak1_re * cs2_im + cz_im * cs2_re;
        cs1_re += b_re * p1_re - absxi * bk;
        cs1_im += b_re * bk + absxi * p1_re;
      }
      y->re = cs1_re * re - cs1_im * im;
      y->im = cs1_re * im + cs1_im * re;
    }
  }
  return nz;
}

} // namespace coder

// End of code generation (casyi.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mtimes.cu
//
// Code generation for function 'mtimes'
//

// Include files
#include "mtimes.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"

// Variable Definitions
static emlrtRTEInfo
    xd_emlrtRTEI{
        158,      // lineNo
        24,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo
    yd_emlrtRTEI{
        140,      // lineNo
        5,        // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

// Function Declarations
static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line);

// Function Definitions
static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

//
//
namespace coder {
namespace internal {
namespace blas {
void mtimes(emxArray_real_T *cpu_A, emxArray_real_T *gpu_A,
            boolean_T *A_outdatedOnGpu, const real_T cpu_B_data[],
            emxArray_real_T *cpu_C, boolean_T *C_outdatedOnCpu,
            emxArray_real_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  real_T *gpu_B_data;
  checkCudaError(mwCudaMalloc(&gpu_B_data, sizeof(real_T)), __FILE__, __LINE__);
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    emxEnsureCapacity_real_T(cpu_C, i, &yd_emlrtRTEI);
    for (i = 0; i < cpu_A->size[0]; i++) {
      cpu_C->data[i] = 0.0;
      *C_outdatedOnGpu = true;
    }
  } else {
    real_T alpha1;
    real_T beta1;
    int32_T i;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    emxEnsureCapacity_real_T(cpu_C, i, &xd_emlrtRTEI);
    alpha1 = 1.0;
    beta1 = 0.0;
    gpuEmxEnsureCapacity_real_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    checkCudaError(hipMemcpy(gpu_B_data, cpu_B_data, sizeof(real_T),
                              hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
    cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            cpu_A->size[0], 1, cpu_A->size[1],
                            (double *)&alpha1, (double *)&gpu_A->data[0],
                            cpu_A->size[0], (double *)&gpu_B_data[0],
                            cpu_A->size[1], (double *)&beta1,
                            (double *)&gpu_C->data[0], cpu_A->size[0]),
                __FILE__, __LINE__);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
  checkCudaError(mwCudaFree(gpu_B_data), __FILE__, __LINE__);
}

} // namespace blas
} // namespace internal
} // namespace coder

// End of code generation (mtimes.cu)

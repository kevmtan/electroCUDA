//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filtfilt_initialize.cu
//
// Code generation for function 'filtfilt_initialize'
//

// Include files
#include "filtfilt_initialize.h"
#include "_coder_filtfilt_mex.h"
#include "filtfilt_data.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void filtfilt_once();

// Function Definitions
static void filtfilt_once()
{
  mex_InitInfAndNan();
  nvtxRangePushA("#initFcn#filtfilt_once#" MW_AT_LOCATION);
  // Initialize GPU by calling hipFree(nullptr)
  nvtxMarkA("#hipFree#" MW_AT_LINE);
  hipFree(nullptr);
  nvtxRangePop();
}

void filtfilt_initialize()
{
  nvtxRangePushA("#fcn#filtfilt_initialize#" MW_AT_LOCATION);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    nvtxMarkA("#filtfilt_once#" MW_AT_LINE);
    filtfilt_once();
  }
  nvtxRangePop();
}

// End of code generation (filtfilt_initialize.cu)

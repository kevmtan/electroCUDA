#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filter.cu
//
// Code generation for function 'filter'
//

// Include files
#include "filter.h"
#include "filtfilt_data.h"
#include "filtfilt_emxutil.h"
#include "filtfilt_mexutil.h"
#include "filtfilt_types.h"
#include "gpufilterImpl.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <algorithm>
#include <cmath>

// Variable Definitions
static emlrtRTEInfo td_emlrtRTEI{
    172,                                                               // lineNo
    13,                                                                // colNo
    "filter",                                                          // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/datafun/filter.m" // pName
};

static emlrtRTEInfo ud_emlrtRTEI{
    1,               // lineNo
    1,               // colNo
    "forGpuCodegen", // fName
    "/usr/local/MATLAB/R2024b/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "+stencil/forGpuCodegen.p" // pName
};

static emlrtRTEInfo vd_emlrtRTEI{
    172,                                                               // lineNo
    9,                                                                 // colNo
    "filter",                                                          // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/datafun/filter.m" // pName
};

static emlrtRTEInfo wd_emlrtRTEI{
    170,                                                               // lineNo
    9,                                                                 // colNo
    "filter",                                                          // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/datafun/filter.m" // pName
};

// Function Declarations
static uint64_T computeNumIters(int32_T ub, int32_T b_ub);

static uint64_T computeNumIters(int32_T ub);

static void cpuEmxAllocateOrResize_real_T(emxArray_real_T *cpu,
                                          boolean_T needsCopy);

static
#ifdef __HIPCC__
    __device__
#endif
        int32_T
        div_s32_device(int32_T numerator, int32_T denominator);

static __global__ void filter_kernel1(const emxArray_real_T a, real_T *a1);

static __global__ void filter_kernel10(const emxArray_real_T b,
                                       const int32_T b_b,
                                       emxArray_real_T kernel1D);

static __global__ void filter_kernel11(const emxArray_real_T a, real_T *a1);

static __global__ void filter_kernel12(const real_T *a1, const int32_T b,
                                       emxArray_real_T b_b);

static __global__ void filter_kernel13(const real_T *a1, const int32_T b,
                                       emxArray_real_T a);

static __global__ void filter_kernel14(emxArray_real_T a);

static __global__ void filter_kernel15(const emxArray_real_T b, const int32_T c,
                                       emxArray_real_T kernel1D);

static __global__ void
filter_kernel16(const int32_T b, emxArray_real_T kernel1D, int32_T b_dim1);

static __global__ void filter_kernel17(const emxArray_real_T x,
                                       const int32_T b_x,
                                       emxArray_real_T input);

static __global__ void filter_kernel18(const emxArray_real_T convOut,
                                       const int32_T nb, const int32_T b,
                                       emxArray_real_T zf);

static __global__ void filter_kernel19(const emxArray_real_T convOut,
                                       const emxArray_real_T a, const int32_T b,
                                       emxArray_real_T zf, int32_T a_dim1,
                                       int32_T x_dim0);

static __global__ void filter_kernel2(const real_T *a1, const int32_T b,
                                      emxArray_real_T b_b);

static __global__ void filter_kernel20(const emxArray_real_T convOut,
                                       const int32_T b, emxArray_real_T y);

static __global__ void filter_kernel21(const emxArray_real_T zi,
                                       const int32_T b,
                                       emxArray_real_T convOut);

static __global__ void filter_kernel22(const emxArray_real_T b,
                                       const int32_T b_b,
                                       emxArray_real_T kernel1D);

static __global__ void filter_kernel23(const emxArray_real_T a, real_T *a1);

static __global__ void filter_kernel24(const real_T *a1, const int32_T b,
                                       emxArray_real_T b_b);

static __global__ void filter_kernel25(const real_T *a1, const int32_T b,
                                       emxArray_real_T a);

static __global__ void filter_kernel26(emxArray_real_T a);

static __global__ void filter_kernel27(const emxArray_real_T b, const int32_T c,
                                       emxArray_real_T kernel1D);

static __global__ void
filter_kernel28(const int32_T b, emxArray_real_T kernel1D, int32_T b_dim1);

static __global__ void filter_kernel29(const emxArray_real_T x,
                                       const int32_T b_x, const int32_T c_x,
                                       emxArray_real_T input,
                                       int32_T input_dim0, int32_T x_dim0);

static __global__ void filter_kernel3(const real_T *a1, const int32_T b,
                                      emxArray_real_T a);

static __global__ void filter_kernel30(const emxArray_real_T zi,
                                       const int32_T b_zi, const int32_T b,
                                       emxArray_real_T convOut,
                                       int32_T convOut_dim0, int32_T zi_dim0);

static __global__ void filter_kernel31(const emxArray_real_T a, const int32_T b,
                                       emxArray_real_T convOut, int32_T x_dim0,
                                       int32_T a_dim1, int32_T convOut_dim0);

static __global__ void
filter_kernel32(const emxArray_real_T convOut, const int32_T nb,
                const int32_T b, const int32_T b_convOut, emxArray_real_T zfIIR,
                int32_T zfIIR_dim0, int32_T convOut_dim0);

static __global__ void filter_kernel33(const emxArray_real_T convOut,
                                       const emxArray_real_T a,
                                       const int32_T b_a, const int32_T b,
                                       emxArray_real_T zfIIR, int32_T a_dim1,
                                       int32_T zfIIR_dim0, int32_T x_dim0,
                                       int32_T convOut_dim0);

static __global__ void filter_kernel34(const emxArray_real_T zfIIR,
                                       const int32_T zfSize_idx_0,
                                       emxArray_real_T zf);

static __global__ void filter_kernel35(const emxArray_real_T convOut,
                                       const int32_T x, const int32_T b_convOut,
                                       emxArray_real_T c_convOut,
                                       int32_T convOut_dim0,
                                       int32_T b_convOut_dim0);

static __global__ void filter_kernel36(const emxArray_real_T convOut,
                                       const int32_T zi, emxArray_real_T y);

static __global__ void filter_kernel37(const emxArray_real_T zi,
                                       const int32_T b_zi, const int32_T b,
                                       emxArray_real_T convOut,
                                       int32_T convOut_dim0);

static __global__ void
filter_kernel38(const emxArray_real_T zi, const emxArray_real_T convOut,
                const int32_T b_zi, const int32_T b_convOut,
                emxArray_real_T c_convOut, int32_T convOut_dim0,
                int32_T b_convOut_dim0, int32_T zi_dim0);

static __global__ void filter_kernel39(const emxArray_real_T convOut,
                                       const int32_T zi_dim0, const int32_T zi,
                                       const int32_T b_zi,
                                       emxArray_real_T b_convOut,
                                       int32_T convOut_dim0);

static __global__ void filter_kernel4(emxArray_real_T a);

static __global__ void filter_kernel40(const emxArray_real_T b,
                                       const int32_T b_b,
                                       emxArray_real_T kernel1D);

static __global__ void filter_kernel41(const emxArray_real_T a, real_T *a1);

static __global__ void filter_kernel42(const real_T *a1, const int32_T b,
                                       emxArray_real_T b_b);

static __global__ void filter_kernel43(const real_T *a1, const int32_T b,
                                       emxArray_real_T a);

static __global__ void filter_kernel44(emxArray_real_T a);

static __global__ void filter_kernel45(const emxArray_real_T b, const int32_T c,
                                       emxArray_real_T kernel1D);

static __global__ void
filter_kernel46(const int32_T b, emxArray_real_T kernel1D, int32_T b_dim1);

static __global__ void filter_kernel47(const emxArray_real_T x,
                                       const int32_T b_x, const int32_T c_x,
                                       emxArray_real_T input,
                                       int32_T input_dim0, int32_T x_dim0);

static __global__ void filter_kernel48(const emxArray_real_T zi,
                                       const int32_T b_zi, const int32_T b,
                                       emxArray_real_T convOut,
                                       int32_T convOut_dim0, int32_T zi_dim0);

static __global__ void filter_kernel49(const emxArray_real_T a, const int32_T b,
                                       emxArray_real_T convOut, int32_T x_dim0,
                                       int32_T a_dim1, int32_T convOut_dim0);

static __global__ void filter_kernel5(const emxArray_real_T b, const int32_T c,
                                      emxArray_real_T kernel1D);

static __global__ void filter_kernel50(const emxArray_real_T convOut,
                                       const int32_T x, const int32_T b_convOut,
                                       emxArray_real_T c_convOut,
                                       int32_T convOut_dim0,
                                       int32_T b_convOut_dim0);

static __global__ void filter_kernel51(const emxArray_real_T convOut,
                                       const int32_T zi, emxArray_real_T y);

static __global__ void filter_kernel52(const emxArray_real_T zi,
                                       const int32_T b_zi, const int32_T b,
                                       emxArray_real_T convOut,
                                       int32_T convOut_dim0);

static __global__ void
filter_kernel53(const emxArray_real_T zi, const emxArray_real_T convOut,
                const int32_T b_zi, const int32_T b_convOut,
                emxArray_real_T c_convOut, int32_T convOut_dim0,
                int32_T b_convOut_dim0, int32_T zi_dim0);

static __global__ void filter_kernel54(const emxArray_real_T convOut,
                                       const int32_T zi_dim0, const int32_T zi,
                                       const int32_T b_zi,
                                       emxArray_real_T b_convOut,
                                       int32_T convOut_dim0);

static __global__ void filter_kernel55(const emxArray_real_T b,
                                       const int32_T b_b,
                                       emxArray_real_T kernel1D);

static __global__ void filter_kernel6(const int32_T b, emxArray_real_T kernel1D,
                                      int32_T b_dim1);

static __global__ void filter_kernel7(const emxArray_real_T x,
                                      const int32_T b_x, emxArray_real_T input);

static __global__ void filter_kernel8(const emxArray_real_T convOut,
                                      const int32_T b, emxArray_real_T y);

static __global__ void filter_kernel9(const emxArray_real_T zi, const int32_T b,
                                      emxArray_real_T convOut);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy);

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu);

static void gpuThrowError(const char_T *file, int32_T b_line);

static
#ifdef __HIPCC__
    __device__
#endif
        real_T
        paddedArrayElem_device(const emxArray_real_T *array, int32_T idx,
                               int32_T b_idx, real_T paddingValue,
                               int32_T array_dim0);

static
#ifdef __HIPCC__
    __device__
#endif
        real_T
        paddedArrayElem_device(const emxArray_real_T *array, int32_T idx,
                               real_T paddingValue, int32_T array_dim0);

static __global__ void stencilfunKernel(
    const emxArray_real_T input, int32_T negPad, real_T paddingValue,
    const emxArray_real_T stencilCapture_workspace_t1, emxArray_real_T output,
    int32_T output_dim0, int32_T output_dim1,
    int32_T c_stencilCapture_workspace_t1_d, int32_T input_dim0);

static __global__ void
stencilfunKernel(const emxArray_real_T input, int32_T negPad,
                 real_T paddingValue,
                 const emxArray_real_T stencilCapture_workspace_t1,
                 emxArray_real_T output, int32_T output_dim0,
                 int32_T c_stencilCapture_workspace_t1_d, int32_T input_dim0);

// Function Definitions
static uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  boolean_T overflow;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  overflow = false;
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
    overflow = (numIters > MAX_uint64_T / static_cast<uint64_T>(b_ub + 1));
  }
  numIters *= n;
  if (overflow) {
    nvtxMarkA("#gpuThrowError#" MW_AT_LINE);
    gpuThrowError(__FILE__, __LINE__);
  }
  nvtxRangePop();
  return numIters;
}

static uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  nvtxRangePop();
  return numIters;
}

static void cpuEmxAllocateOrResize_real_T(emxArray_real_T *cpu,
                                          boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  void *newData;
  nvtxRangePushA("#fcn#cpuEmxAllocateOrResize_real_T#" MW_AT_LOCATION);
  totalSizeCpu = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#cpuEmxAllocateOrResize_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeCpu *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (cpu->allocatedSize < totalSizeCpu) {
    newData =
        emlrtCallocMex(static_cast<uint32_T>(totalSizeCpu), sizeof(real_T));
    needsCopy = (needsCopy && (totalSizeCpu > 0));
    if (needsCopy) {
      std::copy(cpu->data, cpu->data + static_cast<uint32_T>(totalSizeCpu),
                static_cast<real_T *>(newData));
    }
    if (cpu->canFreeData) {
      emlrtFreeMex(cpu->data);
    }
    cpu->data = static_cast<real_T *>(newData);
    cpu->allocatedSize = totalSizeCpu;
    cpu->canFreeData = true;
  }
  nvtxRangePop();
}

static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if (static_cast<int32_T>(numerator < 0) !=
        static_cast<int32_T>(denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static __global__
    __launch_bounds__(32, 1) void filter_kernel1(const emxArray_real_T a,
                                                 real_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel10(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    kernel1D.data[k] = b.data[k];
  }
}

static __global__
    __launch_bounds__(32, 1) void filter_kernel11(const emxArray_real_T a,
                                                  real_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel12(const real_T *a1,
                                                            const int32_T b,
                                                            emxArray_real_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel13(const real_T *a1,
                                                            const int32_T b,
                                                            emxArray_real_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel14(emxArray_real_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel15(
    const emxArray_real_T b, const int32_T c, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    kernel1D.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel16(
    const int32_T b, emxArray_real_T kernel1D, int32_T b_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    kernel1D.data[k + b_dim1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel17(
    const emxArray_real_T x, const int32_T b_x, emxArray_real_T input)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    input.data[k] = x.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel18(
    const emxArray_real_T convOut, const int32_T nb, const int32_T b,
    emxArray_real_T zf)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    zf.data[k] = convOut.data[nb + k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel19(
    const emxArray_real_T convOut, const emxArray_real_T a, const int32_T b,
    emxArray_real_T zf, int32_T a_dim1, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    i = static_cast<int32_T>(idx);
    k = static_cast<int32_T>(
        fmin(static_cast<real_T>(i) + 2.0, static_cast<real_T>(a_dim1)));
    for (int32_T j{0}; j <= k - 2; j++) {
      zf.data[i + 1] += convOut.data[(x_dim0 + i) - j] * a.data[j + 1];
    }
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel2(const real_T *a1,
                                                           const int32_T b,
                                                           emxArray_real_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel20(
    const emxArray_real_T convOut, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = convOut.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel21(
    const emxArray_real_T zi, const int32_T b, emxArray_real_T convOut)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    convOut.data[i] += zi.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel22(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    kernel1D.data[k] = b.data[k];
  }
}

static __global__
    __launch_bounds__(32, 1) void filter_kernel23(const emxArray_real_T a,
                                                  real_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel24(const real_T *a1,
                                                            const int32_T b,
                                                            emxArray_real_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel25(const real_T *a1,
                                                            const int32_T b,
                                                            emxArray_real_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel26(emxArray_real_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel27(
    const emxArray_real_T b, const int32_T c, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    kernel1D.data[i1] = b.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel28(
    const int32_T b, emxArray_real_T kernel1D, int32_T b_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    kernel1D.data[i1 + b_dim1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel29(
    const emxArray_real_T x, const int32_T b_x, const int32_T c_x,
    emxArray_real_T input, int32_T input_dim0, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_x) + 1UL) * (static_cast<uint64_T>(b_x) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_x) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_x) + 1UL));
    input.data[k + input_dim0 * i1] = x.data[k + x_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel3(const real_T *a1,
                                                           const int32_T b,
                                                           emxArray_real_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel30(
    const emxArray_real_T zi, const int32_T b_zi, const int32_T b,
    emxArray_real_T convOut, int32_T convOut_dim0, int32_T zi_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b) + 1UL) * (static_cast<uint64_T>(b_zi) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T m;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_zi) + 1UL));
    m = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_zi) + 1UL));
    convOut.data[i + convOut_dim0 * m] += zi.data[zi_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel31(
    const emxArray_real_T a, const int32_T b, emxArray_real_T convOut,
    int32_T x_dim0, int32_T a_dim1, int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T m;
    m = static_cast<int32_T>(idx);
    i1 = (x_dim0 + a_dim1) - 1;
    for (int32_T i{0}; i < i1; i++) {
      int32_T k;
      k = static_cast<int32_T>(
              fmin(static_cast<real_T>(i) + 1.0, static_cast<real_T>(a_dim1))) -
          1;
      for (int32_T j{0}; j < k; j++) {
        convOut.data[i + convOut_dim0 * m] -=
            convOut.data[((i - j) + convOut_dim0 * m) - 1] * a.data[j + 1];
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel32(
    const emxArray_real_T convOut, const int32_T nb, const int32_T b,
    const int32_T b_convOut, emxArray_real_T zfIIR, int32_T zfIIR_dim0,
    int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_convOut) + 1UL) *
                (static_cast<uint64_T>(b) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b) + 1UL));
    zfIIR.data[k + zfIIR_dim0 * i1] =
        convOut.data[(nb + k) + convOut_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel33(
    const emxArray_real_T convOut, const emxArray_real_T a, const int32_T b_a,
    const int32_T b, emxArray_real_T zfIIR, int32_T a_dim1, int32_T zfIIR_dim0,
    int32_T x_dim0, int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b) + 1UL) * (static_cast<uint64_T>(b_a) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    int32_T m;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_a) + 1UL));
    m = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_a) + 1UL));
    k = static_cast<int32_T>(
            fmin(static_cast<real_T>(i) + 2.0, static_cast<real_T>(a_dim1))) -
        1;
    for (int32_T j{0}; j < k; j++) {
      zfIIR.data[(i + zfIIR_dim0 * m) + 1] +=
          convOut.data[((x_dim0 + i) - j) + convOut_dim0 * m] * a.data[j + 1];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel34(
    const emxArray_real_T zfIIR, const int32_T zfSize_idx_0, emxArray_real_T zf)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(zfSize_idx_0);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    zf.data[i1] = zfIIR.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel35(
    const emxArray_real_T convOut, const int32_T x, const int32_T b_convOut,
    emxArray_real_T c_convOut, int32_T convOut_dim0, int32_T b_convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_convOut) + 1UL) *
                (static_cast<uint64_T>(x) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(x) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(x) + 1UL));
    c_convOut.data[k + convOut_dim0 * i1] =
        convOut.data[k + b_convOut_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel36(
    const emxArray_real_T convOut, const int32_T zi, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(zi);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = convOut.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel37(
    const emxArray_real_T zi, const int32_T b_zi, const int32_T b,
    emxArray_real_T convOut, int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b) + 1UL) * (static_cast<uint64_T>(b_zi) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T m;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_zi) + 1UL));
    m = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_zi) + 1UL));
    convOut.data[i + convOut_dim0 * m] += zi.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel38(
    const emxArray_real_T zi, const emxArray_real_T convOut, const int32_T b_zi,
    const int32_T b_convOut, emxArray_real_T c_convOut, int32_T convOut_dim0,
    int32_T b_convOut_dim0, int32_T zi_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_convOut) + 1UL) *
                (static_cast<uint64_T>(b_zi) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_zi) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_zi) + 1UL));
    c_convOut.data[k + convOut_dim0 * i1] =
        convOut.data[k + b_convOut_dim0 * i1] + zi.data[k + zi_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel39(
    const emxArray_real_T convOut, const int32_T zi_dim0, const int32_T zi,
    const int32_T b_zi, emxArray_real_T b_convOut, int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b_zi) + 1UL) * (static_cast<uint64_T>(zi) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(zi) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(zi) + 1UL));
    b_convOut.data[k + convOut_dim0 * i1] = convOut.data[k + zi_dim0 * i1];
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel4(emxArray_real_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel40(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    kernel1D.data[i1] = b.data[i1];
  }
}

static __global__
    __launch_bounds__(32, 1) void filter_kernel41(const emxArray_real_T a,
                                                  real_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel42(const real_T *a1,
                                                            const int32_T b,
                                                            emxArray_real_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel43(const real_T *a1,
                                                            const int32_T b,
                                                            emxArray_real_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel44(emxArray_real_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel45(
    const emxArray_real_T b, const int32_T c, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    kernel1D.data[i1] = b.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel46(
    const int32_T b, emxArray_real_T kernel1D, int32_T b_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    kernel1D.data[i1 + b_dim1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel47(
    const emxArray_real_T x, const int32_T b_x, const int32_T c_x,
    emxArray_real_T input, int32_T input_dim0, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_x) + 1UL) * (static_cast<uint64_T>(b_x) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_x) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_x) + 1UL));
    input.data[k + input_dim0 * i1] = x.data[k + x_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel48(
    const emxArray_real_T zi, const int32_T b_zi, const int32_T b,
    emxArray_real_T convOut, int32_T convOut_dim0, int32_T zi_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b) + 1UL) * (static_cast<uint64_T>(b_zi) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T m;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_zi) + 1UL));
    m = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_zi) + 1UL));
    convOut.data[i + convOut_dim0 * m] += zi.data[zi_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel49(
    const emxArray_real_T a, const int32_T b, emxArray_real_T convOut,
    int32_T x_dim0, int32_T a_dim1, int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T m;
    m = static_cast<int32_T>(idx);
    i1 = (x_dim0 + a_dim1) - 1;
    for (int32_T i{0}; i < i1; i++) {
      int32_T k;
      k = static_cast<int32_T>(
              fmin(static_cast<real_T>(i) + 1.0, static_cast<real_T>(a_dim1))) -
          1;
      for (int32_T j{0}; j < k; j++) {
        convOut.data[i + convOut_dim0 * m] -=
            convOut.data[((i - j) + convOut_dim0 * m) - 1] * a.data[j + 1];
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel5(
    const emxArray_real_T b, const int32_T c, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    kernel1D.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel50(
    const emxArray_real_T convOut, const int32_T x, const int32_T b_convOut,
    emxArray_real_T c_convOut, int32_T convOut_dim0, int32_T b_convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_convOut) + 1UL) *
                (static_cast<uint64_T>(x) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(x) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(x) + 1UL));
    c_convOut.data[k + convOut_dim0 * i1] =
        convOut.data[k + b_convOut_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel51(
    const emxArray_real_T convOut, const int32_T zi, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(zi);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = convOut.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel52(
    const emxArray_real_T zi, const int32_T b_zi, const int32_T b,
    emxArray_real_T convOut, int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b) + 1UL) * (static_cast<uint64_T>(b_zi) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T m;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_zi) + 1UL));
    m = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b_zi) + 1UL));
    convOut.data[i + convOut_dim0 * m] += zi.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel53(
    const emxArray_real_T zi, const emxArray_real_T convOut, const int32_T b_zi,
    const int32_T b_convOut, emxArray_real_T c_convOut, int32_T convOut_dim0,
    int32_T b_convOut_dim0, int32_T zi_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_convOut) + 1UL) *
                (static_cast<uint64_T>(b_zi) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_zi) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_zi) + 1UL));
    c_convOut.data[k + convOut_dim0 * i1] =
        convOut.data[k + b_convOut_dim0 * i1] + zi.data[k + zi_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel54(
    const emxArray_real_T convOut, const int32_T zi_dim0, const int32_T zi,
    const int32_T b_zi, emxArray_real_T b_convOut, int32_T convOut_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b_zi) + 1UL) * (static_cast<uint64_T>(zi) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(zi) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(zi) + 1UL));
    b_convOut.data[k + convOut_dim0 * i1] = convOut.data[k + zi_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel55(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T kernel1D)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    kernel1D.data[i1] = b.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel6(
    const int32_T b, emxArray_real_T kernel1D, int32_T b_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    kernel1D.data[k + b_dim1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel7(
    const emxArray_real_T x, const int32_T b_x, emxArray_real_T input)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    input.data[k] = x.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel8(
    const emxArray_real_T convOut, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = convOut.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel9(
    const emxArray_real_T zi, const int32_T b, emxArray_real_T convOut)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    convOut.data[k] += zi.data[k];
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

static void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                        const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (cpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(gpu->data, cpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(real_T),
                   hipMemcpyHostToDevice),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

static void gpuThrowError(const char_T *file, int32_T b_line)
{
  emlrtRTEInfo rtInfo;
  nvtxRangePushA("#fcn#gpuThrowError#" MW_AT_LOCATION);
  rtInfo.lineNo = b_line;
  rtInfo.colNo = 0;
  rtInfo.fName = "";
  rtInfo.pName = file;
  emlrtCUDAError(
      0U, (char_T *)"_",
      (char_T
           *)"Unable to launch kernel. Loop nest contains too many iterations.",
      &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

static __device__ real_T paddedArrayElem_device(const emxArray_real_T *array,
                                                int32_T idx, int32_T b_idx,
                                                real_T paddingValue,
                                                int32_T array_dim0)
{
  real_T output;
  if ((idx >= 0) && (idx < array_dim0)) {
    output = array->data[idx + array_dim0 * b_idx];
  } else {
    output = paddingValue;
  }
  return output;
}

static __device__ real_T paddedArrayElem_device(const emxArray_real_T *array,
                                                int32_T idx,
                                                real_T paddingValue,
                                                int32_T array_dim0)
{
  real_T output;
  if ((idx >= 0) && (idx < array_dim0)) {
    output = array->data[idx];
  } else {
    output = paddingValue;
  }
  return output;
}

static __global__ __launch_bounds__(1024, 1) void stencilfunKernel(
    const emxArray_real_T input, int32_T negPad, real_T paddingValue,
    const emxArray_real_T stencilCapture_workspace_t1, emxArray_real_T output,
    int32_T output_dim0, int32_T output_dim1,
    int32_T c_stencilCapture_workspace_t1_d, int32_T input_dim0)
{
  int32_T workItemLocalIdTmp;
  int32_T workItemLocalOutputElemDimIdx;
  workItemLocalIdTmp = static_cast<int32_T>(mwGetThreadIndexWithinBlock());
  workItemLocalOutputElemDimIdx = workItemLocalIdTmp % 32;
  workItemLocalIdTmp /= 32;
  for (int32_T quasiWorkGroupId{static_cast<int32_T>(mwGetBlockIndex())};
       quasiWorkGroupId <
       static_cast<int32_T>(
           (static_cast<uint32_T>(output_dim0) >> 5) +
           static_cast<uint32_T>(static_cast<int32_T>(
               (static_cast<uint32_T>(output_dim0) & 31U) != 0U))) *
           static_cast<int32_T>(
               (static_cast<uint32_T>(output_dim1) >> 5) +
               static_cast<uint32_T>(static_cast<int32_T>(
                   (static_cast<uint32_T>(output_dim1) & 31U) != 0U)));
       quasiWorkGroupId += static_cast<int32_T>(mwGetBlocksPerGrid())) {
    int32_T numQuasiWorkGroupsAlongDim;
    int32_T workItemGlobalOutputElemDimIdx;
    numQuasiWorkGroupsAlongDim = static_cast<int32_T>(
        (static_cast<uint32_T>(output_dim0) >> 5) +
        static_cast<uint32_T>(static_cast<int32_T>(
            (static_cast<uint32_T>(output_dim0) & 31U) != 0U)));
    workItemGlobalOutputElemDimIdx =
        workItemLocalOutputElemDimIdx +
        ((quasiWorkGroupId % numQuasiWorkGroupsAlongDim) << 5);
    numQuasiWorkGroupsAlongDim =
        workItemLocalIdTmp +
        (div_s32_device(quasiWorkGroupId, numQuasiWorkGroupsAlongDim) << 5);
    if ((workItemGlobalOutputElemDimIdx < output_dim0) &&
        (numQuasiWorkGroupsAlongDim < output_dim1)) {
      real_T varargout_1;
      varargout_1 = 0.0;
      for (int32_T m{0}; m < c_stencilCapture_workspace_t1_d; m++) {
        real_T varargin_1;
        varargin_1 = paddedArrayElem_device(
            &input, (workItemGlobalOutputElemDimIdx + m) - negPad,
            numQuasiWorkGroupsAlongDim, paddingValue, input_dim0);
        varargout_1 +=
            varargin_1 * stencilCapture_workspace_t1
                             .data[(c_stencilCapture_workspace_t1_d - m) - 1];
      }
      output.data[workItemGlobalOutputElemDimIdx +
                  output_dim0 * numQuasiWorkGroupsAlongDim] = varargout_1;
    }
  }
}

static __global__ __launch_bounds__(32, 1) void stencilfunKernel(
    const emxArray_real_T input, int32_T negPad, real_T paddingValue,
    const emxArray_real_T stencilCapture_workspace_t1, emxArray_real_T output,
    int32_T output_dim0, int32_T c_stencilCapture_workspace_t1_d,
    int32_T input_dim0)
{
  int32_T workItemLocalIdTmp;
  workItemLocalIdTmp = static_cast<int32_T>(mwGetThreadIndexWithinBlock());
  for (int32_T quasiWorkGroupId{static_cast<int32_T>(mwGetBlockIndex())};
       quasiWorkGroupId <
       static_cast<int32_T>(
           (static_cast<uint32_T>(output_dim0) >> 5) +
           static_cast<uint32_T>(static_cast<int32_T>(
               (static_cast<uint32_T>(output_dim0) & 31U) != 0U)));
       quasiWorkGroupId += static_cast<int32_T>(mwGetBlocksPerGrid())) {
    int32_T workItemGlobalOutputElemDimIdx;
    workItemGlobalOutputElemDimIdx =
        workItemLocalIdTmp + (quasiWorkGroupId << 5);
    if (workItemGlobalOutputElemDimIdx < output_dim0) {
      real_T varargout_1;
      varargout_1 = 0.0;
      for (int32_T m{0}; m < c_stencilCapture_workspace_t1_d; m++) {
        real_T varargin_1;
        varargin_1 = paddedArrayElem_device(
            &input, (workItemGlobalOutputElemDimIdx + m) - negPad, paddingValue,
            input_dim0);
        varargout_1 +=
            varargin_1 * stencilCapture_workspace_t1
                             .data[(c_stencilCapture_workspace_t1_d - m) - 1];
      }
      output.data[workItemGlobalOutputElemDimIdx] = varargout_1;
    }
  }
}

//
//
namespace coder {
void b_filter(emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
              emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
              emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
              emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
              emxArray_real_T *cpu_x, emxArray_real_T *gpu_x,
              boolean_T *x_outdatedOnGpu, emxArray_real_T *cpu_zi,
              boolean_T *zi_outdatedOnCpu, emxArray_real_T *gpu_zi,
              boolean_T *zi_outdatedOnGpu, emxArray_real_T *cpu_y,
              boolean_T *y_outdatedOnCpu, emxArray_real_T *gpu_y,
              boolean_T *y_outdatedOnGpu, emxArray_real_T *cpu_zf,
              boolean_T *zf_outdatedOnCpu, emxArray_real_T *gpu_zf,
              boolean_T *zf_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T gpu_convOut;
  emxArray_real_T gpu_input;
  emxArray_real_T gpu_kernel1D;
  emxArray_real_T *cpu_convOut;
  emxArray_real_T *cpu_input;
  emxArray_real_T *cpu_kernel1D;
  real_T *gpu_a1;
  int32_T na;
  int32_T nb;
  int32_T zfSize_idx_0;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#b_filter#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_kernel1D);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a1, 8UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[1];
  nb = cpu_b->size[1];
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
  }
  nvtxMarkA("#filter_kernel11#" MW_AT_LINE);
  filter_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
  cpuEmxAllocateOrResize_real_T(cpu_a, !*a_outdatedOnCpu);
  if (*a_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0])) &&
      (!(cpu_a->data[0] == 0.0)) && (cpu_a->data[0] != 1.0)) {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel12#" MW_AT_LINE);
      filter_kernel12<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel13#" MW_AT_LINE);
      filter_kernel13<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
    }
    nvtxMarkA("#filter_kernel14#" MW_AT_LINE);
    filter_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
    *a_outdatedOnCpu = true;
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_kernel1D, 1, &sd_emlrtRTEI, true);
  if (cpu_a->size[1] > cpu_b->size[1]) {
    cpu_kernel1D->size[0] = cpu_a->size[1];
    na = cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel15#" MW_AT_LINE);
      filter_kernel15<<<grid, block>>>(*gpu_b, na - 1, gpu_kernel1D);
    }
    na = cpu_a->size[1] - cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel16#" MW_AT_LINE);
      filter_kernel16<<<grid, block>>>(na - 1, gpu_kernel1D, cpu_b->size[1U]);
    }
  } else {
    cpu_kernel1D->size[0] = cpu_b->size[1];
    na = cpu_b->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel22#" MW_AT_LINE);
      filter_kernel22<<<grid, block>>>(*gpu_b, na, gpu_kernel1D);
    }
  }
  zfSize_idx_0 = cpu_kernel1D->size[0] - 1;
  if (cpu_x->size[0] == 0) {
    *y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
    *zf_outdatedOnCpu = false;
    na = cpu_zf->size[0];
    cpu_zf->size[0] = cpu_kernel1D->size[0] - 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_zf, na, &sd_emlrtRTEI);
  } else {
    dim3 b_grid;
    int32_T i;
    uint32_T varargin_3_idx_0;
    boolean_T convOut_outdatedOnCpu;
    boolean_T convOut_outdatedOnGpu;
    varargin_3_idx_0 = static_cast<uint32_T>(cpu_kernel1D->size[0]);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_input, 1, &wd_emlrtRTEI, true);
    cpu_input->size[0] = cpu_x->size[0];
    na = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_input, &gpu_input, true);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel17#" MW_AT_LINE);
      filter_kernel17<<<grid, block>>>(*gpu_x, na, gpu_input);
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_convOut, 1, &sd_emlrtRTEI, true);
    na = cpu_convOut->size[0];
    cpu_convOut->size[0] = (cpu_x->size[0] + cpu_kernel1D->size[0]) - 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_convOut, na, &ud_emlrtRTEI);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    b_grid =
        dim3((static_cast<uint32_T>(cpu_convOut->size[0]) >> 5) +
                 ((static_cast<uint32_T>(cpu_convOut->size[0]) & 31U) != 0U),
             1U, 1U);
    validLaunchParams = mwValidateLaunchParameters(b_grid, dim3(32U, 1U, 1U));
    if (validLaunchParams) {
      nvtxMarkA("#stencilfunKernel#" MW_AT_LINE);
      stencilfunKernel<<<b_grid, dim3(32U, 1U, 1U)>>>(
          gpu_input, static_cast<int32_T>(varargin_3_idx_0) - 1, 0.0,
          gpu_kernel1D, gpu_convOut, cpu_convOut->size[0U],
          cpu_kernel1D->size[0U], cpu_input->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    convOut_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_input);
    if (cpu_zi->size[0] != 0) {
      if (cpu_zi->size[0] == 1) {
        nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
        cpuEmxAllocateOrResize_real_T(cpu_zi, !*zi_outdatedOnCpu);
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        cpu_convOut->data[0] += cpu_zi->data[0];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      } else {
        na = cpu_zi->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel21#" MW_AT_LINE);
          filter_kernel21<<<grid, block>>>(*gpu_zi, na - 1, gpu_convOut);
        }
      }
    }
    na = cpu_x->size[0] + cpu_a->size[1];
    profileLoopStart("b_filter_loop_0", __LINE__, (na - 2) + 1, "");
    for (i = 0; i <= na - 2; i++) {
      nb = static_cast<int32_T>(std::fmin(static_cast<real_T>(i) + 1.0,
                                          static_cast<real_T>(cpu_a->size[1])));
      profileLoopStart("b_filter_loop_1", __LINE__, (nb - 2) + 1, "");
      for (int32_T j{0}; j <= nb - 2; j++) {
        if (convOut_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_convOut, &gpu_convOut);
        }
        if (*a_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
        }
        *a_outdatedOnCpu = false;
        cpu_convOut->data[i] -=
            cpu_convOut->data[(i - j) - 1] * cpu_a->data[j + 1];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    if (static_cast<uint32_T>(cpu_x->size[0]) + 1U >
        static_cast<uint32_T>(cpu_convOut->size[0])) {
      nb = 0;
      i = 0;
    } else {
      nb = cpu_x->size[0];
      i = cpu_convOut->size[0];
    }
    na = cpu_zf->size[0];
    cpu_zf->size[0] = i - nb;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_zf, na, &sd_emlrtRTEI);
    na = (i - nb) - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut,
                                  !convOut_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_zf, gpu_zf, true);
    if (convOut_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel18#" MW_AT_LINE);
      filter_kernel18<<<grid, block>>>(gpu_convOut, nb, na, *gpu_zf);
    }
    na = cpu_a->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 3), &grid, &block,
                            2147483647U);
    *x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel19#" MW_AT_LINE);
      filter_kernel19<<<grid, block>>>(gpu_convOut, *gpu_a, na - 3, *gpu_zf,
                                       cpu_a->size[1U], cpu_x->size[0U]);
    }
    *zf_outdatedOnCpu = true;
    na = cpu_zf->size[0];
    cpu_zf->size[0] = zfSize_idx_0;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_zf, na, &sd_emlrtRTEI);
    na = cpu_y->size[0];
    cpu_y->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, na, &sd_emlrtRTEI);
    na = cpu_x->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_y, gpu_y, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel20#" MW_AT_LINE);
      filter_kernel20<<<grid, block>>>(gpu_convOut, na - 1, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_convOut);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_kernel1D);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_kernel1D);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_convOut);
  *b_outdatedOnGpu = false;
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
  *zf_outdatedOnGpu = false;
  nvtxRangePop();
}

//
//
void c_filter(emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
              emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
              emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
              emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
              emxArray_real_T *cpu_x, emxArray_real_T *gpu_x,
              boolean_T *x_outdatedOnGpu, emxArray_real_T *cpu_zi,
              boolean_T *zi_outdatedOnCpu, emxArray_real_T *gpu_zi,
              boolean_T *zi_outdatedOnGpu, emxArray_real_T *cpu_y,
              boolean_T *y_outdatedOnCpu, emxArray_real_T *gpu_y,
              boolean_T *y_outdatedOnGpu, emxArray_real_T *cpu_zf,
              boolean_T *zf_outdatedOnCpu, emxArray_real_T *gpu_zf,
              boolean_T *zf_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T b_gpu_convOut;
  emxArray_real_T c_gpu_convOut;
  emxArray_real_T gpu_convOut;
  emxArray_real_T gpu_input;
  emxArray_real_T gpu_kernel1D;
  emxArray_real_T gpu_zfIIR;
  emxArray_real_T *b_cpu_convOut;
  emxArray_real_T *c_cpu_convOut;
  emxArray_real_T *cpu_convOut;
  emxArray_real_T *cpu_input;
  emxArray_real_T *cpu_kernel1D;
  emxArray_real_T *cpu_zfIIR;
  real_T *gpu_a1;
  int32_T b;
  int32_T na;
  int32_T nb;
  int32_T zfSize_idx_0;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#c_filter#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_zfIIR);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_kernel1D);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a1, 8UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[1];
  nb = cpu_b->size[1];
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
  }
  nvtxMarkA("#filter_kernel23#" MW_AT_LINE);
  filter_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
  cpuEmxAllocateOrResize_real_T(cpu_a, !*a_outdatedOnCpu);
  if (*a_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0])) &&
      (!(cpu_a->data[0] == 0.0)) && (cpu_a->data[0] != 1.0)) {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel24#" MW_AT_LINE);
      filter_kernel24<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel25#" MW_AT_LINE);
      filter_kernel25<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
    }
    nvtxMarkA("#filter_kernel26#" MW_AT_LINE);
    filter_kernel26<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
    *a_outdatedOnCpu = true;
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_kernel1D, 1, &sd_emlrtRTEI, true);
  if (cpu_a->size[1] > cpu_b->size[1]) {
    cpu_kernel1D->size[0] = cpu_a->size[1];
    na = cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel27#" MW_AT_LINE);
      filter_kernel27<<<grid, block>>>(*gpu_b, na - 1, gpu_kernel1D);
    }
    na = cpu_a->size[1] - cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel28#" MW_AT_LINE);
      filter_kernel28<<<grid, block>>>(na - 1, gpu_kernel1D, cpu_b->size[1U]);
    }
  } else {
    cpu_kernel1D->size[0] = cpu_b->size[1];
    b = cpu_b->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(b), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel40#" MW_AT_LINE);
      filter_kernel40<<<grid, block>>>(*gpu_b, b, gpu_kernel1D);
    }
  }
  zfSize_idx_0 = cpu_kernel1D->size[0] - 1;
  if ((cpu_x->size[0] == 0) || (cpu_x->size[1] == 0)) {
    uint32_T varargin_3[2];
    profileLoopStart("c_filter_loop_0", __LINE__, 1 + 1, "");
    for (b = 0; b < 2; b++) {
      varargin_3[b] = static_cast<uint32_T>(cpu_x->size[b]);
    }
    profileLoopEnd();
    *y_outdatedOnCpu = false;
    na = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = static_cast<int32_T>(varargin_3[0]);
    cpu_y->size[1] = static_cast<int32_T>(varargin_3[1]);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, na, &sd_emlrtRTEI);
    *zf_outdatedOnCpu = false;
    na = cpu_zf->size[0] * cpu_zf->size[1];
    cpu_zf->size[0] = zfSize_idx_0;
    cpu_zf->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_zf, na, &sd_emlrtRTEI);
  } else {
    dim3 b_grid;
    uint64_T numIters;
    int32_T zi[2];
    int32_T convOut;
    uint32_T varargin_3[2];
    boolean_T convOut_outdatedOnGpu;
    varargin_3[0] = static_cast<uint32_T>(cpu_kernel1D->size[0]);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_input, 2, &sd_emlrtRTEI, true);
    cpu_input->size[0] = cpu_x->size[0];
    cpu_input->size[1] = cpu_x->size[1];
    na = cpu_x->size[1] - 1;
    b = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    numIters = computeNumIters(na, b);
    mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_input, &gpu_input, true);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel29#" MW_AT_LINE);
      filter_kernel29<<<grid, block>>>(*gpu_x, b, na, gpu_input,
                                       cpu_input->size[0U], cpu_x->size[0U]);
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_convOut, 2, &sd_emlrtRTEI, true);
    na = cpu_convOut->size[0] * cpu_convOut->size[1];
    cpu_convOut->size[0] = (cpu_x->size[0] + cpu_kernel1D->size[0]) - 1;
    cpu_convOut->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_convOut, na, &ud_emlrtRTEI);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    b_grid = dim3(
        static_cast<uint32_T>(
            static_cast<int32_T>(
                (static_cast<uint32_T>(cpu_convOut->size[0]) >> 5) +
                ((static_cast<uint32_T>(cpu_convOut->size[0]) & 31U) != 0U)) *
            static_cast<int32_T>(
                (static_cast<uint32_T>(cpu_convOut->size[1]) >> 5) +
                ((static_cast<uint32_T>(cpu_convOut->size[1]) & 31U) != 0U))),
        1U, 1U);
    validLaunchParams = mwValidateLaunchParameters(b_grid, dim3(1024U, 1U, 1U));
    if (validLaunchParams) {
      nvtxMarkA("#stencilfunKernel#" MW_AT_LINE);
      stencilfunKernel<<<b_grid, dim3(1024U, 1U, 1U)>>>(
          gpu_input, static_cast<int32_T>(varargin_3[0]) - 1, 0.0, gpu_kernel1D,
          gpu_convOut, cpu_convOut->size[0U], cpu_convOut->size[1U],
          cpu_kernel1D->size[0U], cpu_input->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_input);
    if ((cpu_zi->size[0] != 0) && (cpu_zi->size[1] != 0)) {
      if (cpu_zi->size[0] == 1) {
        na = cpu_convOut->size[1];
        b = cpu_zi->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(na - 1, b);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel30#" MW_AT_LINE);
          filter_kernel30<<<grid, block>>>(*gpu_zi, b, na - 1, gpu_convOut,
                                           cpu_convOut->size[0U],
                                           cpu_zi->size[0U]);
        }
      } else if (cpu_zi->size[1] == 1) {
        na = cpu_convOut->size[1];
        b = cpu_zi->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(na - 1, b);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel37#" MW_AT_LINE);
          filter_kernel37<<<grid, block>>>(*gpu_zi, b, na - 1, gpu_convOut,
                                           cpu_convOut->size[0U]);
        }
      } else if (cpu_convOut->size[1] == cpu_zi->size[1]) {
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&b_cpu_convOut, 2, &sd_emlrtRTEI, true);
        b_cpu_convOut->size[0] = cpu_zi->size[0];
        b_cpu_convOut->size[1] = cpu_convOut->size[1];
        convOut = cpu_convOut->size[1] - 1;
        b = cpu_zi->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(convOut, b);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_convOut, &c_gpu_convOut, true);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel38#" MW_AT_LINE);
          filter_kernel38<<<grid, block>>>(
              *gpu_zi, gpu_convOut, b, convOut, c_gpu_convOut,
              b_cpu_convOut->size[0U], cpu_convOut->size[0U], cpu_zi->size[0U]);
        }
        zi[0] = cpu_zi->size[0];
        zi[1] = cpu_convOut->size[1];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(zi[1] - 1, zi[0] - 1);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel39#" MW_AT_LINE);
          filter_kernel39<<<grid, block>>>(c_gpu_convOut, zi[0], zi[0] - 1,
                                           zi[1] - 1, gpu_convOut,
                                           cpu_convOut->size[0U]);
        }
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&b_cpu_convOut);
      } else {
        nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
        cpuEmxAllocateOrResize_real_T(cpu_zi, !*zi_outdatedOnCpu);
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        nvtxMarkA("#binary_expand_op_2#" MW_AT_LINE);
        binary_expand_op_2(cpu_convOut, cpu_zi);
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      }
    }
    na = cpu_convOut->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut,
                                  !convOut_outdatedOnGpu);
    }
    if (convOut_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel31#" MW_AT_LINE);
      filter_kernel31<<<grid, block>>>(*gpu_a, na - 1, gpu_convOut,
                                       cpu_x->size[0U], cpu_a->size[1U],
                                       cpu_convOut->size[0U]);
    }
    if (static_cast<uint32_T>(cpu_x->size[0]) + 1U >
        static_cast<uint32_T>(cpu_convOut->size[0])) {
      nb = 0;
      b = 0;
    } else {
      nb = cpu_x->size[0];
      b = cpu_convOut->size[0];
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_zfIIR, 2, &sd_emlrtRTEI, true);
    cpu_zfIIR->size[0] = b - nb;
    cpu_zfIIR->size[1] = cpu_convOut->size[1];
    convOut = cpu_convOut->size[1] - 1;
    b = (b - nb) - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    numIters = computeNumIters(convOut, b);
    mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_zfIIR, &gpu_zfIIR, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel32#" MW_AT_LINE);
      filter_kernel32<<<grid, block>>>(gpu_convOut, nb, b, convOut, gpu_zfIIR,
                                       cpu_zfIIR->size[0U],
                                       cpu_convOut->size[0U]);
    }
    na = cpu_convOut->size[1];
    b = cpu_a->size[1] - 3;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    numIters = computeNumIters(na - 1, b);
    mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
    *x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel33#" MW_AT_LINE);
      filter_kernel33<<<grid, block>>>(
          gpu_convOut, *gpu_a, b, na - 1, gpu_zfIIR, cpu_a->size[1U],
          cpu_zfIIR->size[0U], cpu_x->size[0U], cpu_convOut->size[0U]);
    }
    na = cpu_zf->size[0] * cpu_zf->size[1];
    cpu_zf->size[0] = zfSize_idx_0;
    cpu_zf->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_zf, na, &sd_emlrtRTEI);
    zfSize_idx_0 = zfSize_idx_0 * cpu_x->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(zfSize_idx_0), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_zf, gpu_zf, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel34#" MW_AT_LINE);
      filter_kernel34<<<grid, block>>>(gpu_zfIIR, zfSize_idx_0, *gpu_zf);
    }
    *zf_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_zfIIR);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_cpu_convOut, 2, &sd_emlrtRTEI, true);
    c_cpu_convOut->size[0] = cpu_x->size[0];
    c_cpu_convOut->size[1] = cpu_convOut->size[1];
    convOut = cpu_convOut->size[1] - 1;
    na = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    numIters = computeNumIters(convOut, na);
    mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(c_cpu_convOut, &b_gpu_convOut, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel35#" MW_AT_LINE);
      filter_kernel35<<<grid, block>>>(gpu_convOut, na, convOut, b_gpu_convOut,
                                       c_cpu_convOut->size[0U],
                                       cpu_convOut->size[0U]);
    }
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_convOut);
    zi[0] = cpu_x->size[0];
    zi[1] = cpu_x->size[1];
    na = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = cpu_x->size[0];
    cpu_y->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, na, &sd_emlrtRTEI);
    b = zi[0] * zi[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(b), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_y, gpu_y, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel36#" MW_AT_LINE);
      filter_kernel36<<<grid, block>>>(b_gpu_convOut, b, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_cpu_convOut);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_kernel1D);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_kernel1D);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_convOut);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_zfIIR);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_convOut);
  *b_outdatedOnGpu = false;
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
  *zf_outdatedOnGpu = false;
  nvtxRangePop();
}

//
//
void d_filter(emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
              emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
              emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
              emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
              emxArray_real_T *cpu_x, emxArray_real_T *gpu_x,
              boolean_T *x_outdatedOnGpu, emxArray_real_T *cpu_zi,
              boolean_T *zi_outdatedOnCpu, emxArray_real_T *gpu_zi,
              boolean_T *zi_outdatedOnGpu, emxArray_real_T *cpu_y,
              boolean_T *y_outdatedOnCpu, emxArray_real_T *gpu_y,
              boolean_T *y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T b_gpu_convOut;
  emxArray_real_T c_gpu_convOut;
  emxArray_real_T gpu_convOut;
  emxArray_real_T gpu_input;
  emxArray_real_T gpu_kernel1D;
  emxArray_real_T *b_cpu_convOut;
  emxArray_real_T *c_cpu_convOut;
  emxArray_real_T *cpu_convOut;
  emxArray_real_T *cpu_input;
  emxArray_real_T *cpu_kernel1D;
  real_T *gpu_a1;
  int32_T na;
  int32_T nb;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#d_filter#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_kernel1D);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a1, 8UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[1];
  nb = cpu_b->size[1];
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
  }
  nvtxMarkA("#filter_kernel41#" MW_AT_LINE);
  filter_kernel41<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
  cpuEmxAllocateOrResize_real_T(cpu_a, !*a_outdatedOnCpu);
  if (*a_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0])) &&
      (!(cpu_a->data[0] == 0.0)) && (cpu_a->data[0] != 1.0)) {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel42#" MW_AT_LINE);
      filter_kernel42<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel43#" MW_AT_LINE);
      filter_kernel43<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
    }
    nvtxMarkA("#filter_kernel44#" MW_AT_LINE);
    filter_kernel44<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
    *a_outdatedOnCpu = true;
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_kernel1D, 1, &sd_emlrtRTEI, true);
  if (cpu_a->size[1] > cpu_b->size[1]) {
    cpu_kernel1D->size[0] = cpu_a->size[1];
    na = cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel45#" MW_AT_LINE);
      filter_kernel45<<<grid, block>>>(*gpu_b, na - 1, gpu_kernel1D);
    }
    na = cpu_a->size[1] - cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel46#" MW_AT_LINE);
      filter_kernel46<<<grid, block>>>(na - 1, gpu_kernel1D, cpu_b->size[1U]);
    }
  } else {
    cpu_kernel1D->size[0] = cpu_b->size[1];
    nb = cpu_b->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel55#" MW_AT_LINE);
      filter_kernel55<<<grid, block>>>(*gpu_b, nb, gpu_kernel1D);
    }
  }
  if ((cpu_x->size[0] == 0) || (cpu_x->size[1] == 0)) {
    uint32_T varargin_3[2];
    profileLoopStart("d_filter_loop_0", __LINE__, 1 + 1, "");
    for (nb = 0; nb < 2; nb++) {
      varargin_3[nb] = static_cast<uint32_T>(cpu_x->size[nb]);
    }
    profileLoopEnd();
    *y_outdatedOnCpu = false;
    na = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = static_cast<int32_T>(varargin_3[0]);
    cpu_y->size[1] = static_cast<int32_T>(varargin_3[1]);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, na, &sd_emlrtRTEI);
  } else {
    dim3 b_grid;
    uint64_T numIters;
    int32_T zi[2];
    int32_T b_zi;
    uint32_T varargin_3[2];
    boolean_T convOut_outdatedOnGpu;
    varargin_3[0] = static_cast<uint32_T>(cpu_kernel1D->size[0]);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_input, 2, &sd_emlrtRTEI, true);
    cpu_input->size[0] = cpu_x->size[0];
    cpu_input->size[1] = cpu_x->size[1];
    na = cpu_x->size[1] - 1;
    nb = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    numIters = computeNumIters(na, nb);
    mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_input, &gpu_input, true);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel47#" MW_AT_LINE);
      filter_kernel47<<<grid, block>>>(*gpu_x, nb, na, gpu_input,
                                       cpu_input->size[0U], cpu_x->size[0U]);
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_convOut, 2, &sd_emlrtRTEI, true);
    na = cpu_convOut->size[0] * cpu_convOut->size[1];
    cpu_convOut->size[0] = (cpu_x->size[0] + cpu_kernel1D->size[0]) - 1;
    cpu_convOut->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_convOut, na, &ud_emlrtRTEI);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    b_grid = dim3(
        static_cast<uint32_T>(
            static_cast<int32_T>(
                (static_cast<uint32_T>(cpu_convOut->size[0]) >> 5) +
                ((static_cast<uint32_T>(cpu_convOut->size[0]) & 31U) != 0U)) *
            static_cast<int32_T>(
                (static_cast<uint32_T>(cpu_convOut->size[1]) >> 5) +
                ((static_cast<uint32_T>(cpu_convOut->size[1]) & 31U) != 0U))),
        1U, 1U);
    validLaunchParams = mwValidateLaunchParameters(b_grid, dim3(1024U, 1U, 1U));
    if (validLaunchParams) {
      nvtxMarkA("#stencilfunKernel#" MW_AT_LINE);
      stencilfunKernel<<<b_grid, dim3(1024U, 1U, 1U)>>>(
          gpu_input, static_cast<int32_T>(varargin_3[0]) - 1, 0.0, gpu_kernel1D,
          gpu_convOut, cpu_convOut->size[0U], cpu_convOut->size[1U],
          cpu_kernel1D->size[0U], cpu_input->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_input);
    if ((cpu_zi->size[0] != 0) && (cpu_zi->size[1] != 0)) {
      if (cpu_zi->size[0] == 1) {
        na = cpu_convOut->size[1];
        b_zi = cpu_zi->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(na - 1, b_zi);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel48#" MW_AT_LINE);
          filter_kernel48<<<grid, block>>>(*gpu_zi, b_zi, na - 1, gpu_convOut,
                                           cpu_convOut->size[0U],
                                           cpu_zi->size[0U]);
        }
      } else if (cpu_zi->size[1] == 1) {
        na = cpu_convOut->size[1];
        b_zi = cpu_zi->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(na - 1, b_zi);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel52#" MW_AT_LINE);
          filter_kernel52<<<grid, block>>>(*gpu_zi, b_zi, na - 1, gpu_convOut,
                                           cpu_convOut->size[0U]);
        }
      } else if (cpu_convOut->size[1] == cpu_zi->size[1]) {
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&b_cpu_convOut, 2, &sd_emlrtRTEI, true);
        b_cpu_convOut->size[0] = cpu_zi->size[0];
        b_cpu_convOut->size[1] = cpu_convOut->size[1];
        nb = cpu_convOut->size[1] - 1;
        b_zi = cpu_zi->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(nb, b_zi);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_convOut, &c_gpu_convOut, true);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel53#" MW_AT_LINE);
          filter_kernel53<<<grid, block>>>(
              *gpu_zi, gpu_convOut, b_zi, nb, c_gpu_convOut,
              b_cpu_convOut->size[0U], cpu_convOut->size[0U], cpu_zi->size[0U]);
        }
        zi[0] = cpu_zi->size[0];
        zi[1] = cpu_convOut->size[1];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        numIters = computeNumIters(zi[1] - 1, zi[0] - 1);
        mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel54#" MW_AT_LINE);
          filter_kernel54<<<grid, block>>>(c_gpu_convOut, zi[0], zi[0] - 1,
                                           zi[1] - 1, gpu_convOut,
                                           cpu_convOut->size[0U]);
        }
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&b_cpu_convOut);
      } else {
        nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
        cpuEmxAllocateOrResize_real_T(cpu_zi, !*zi_outdatedOnCpu);
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        nvtxMarkA("#binary_expand_op_2#" MW_AT_LINE);
        binary_expand_op_2(cpu_convOut, cpu_zi);
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      }
    }
    na = cpu_convOut->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut,
                                  !convOut_outdatedOnGpu);
    }
    *x_outdatedOnGpu = false;
    if (convOut_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel49#" MW_AT_LINE);
      filter_kernel49<<<grid, block>>>(*gpu_a, na - 1, gpu_convOut,
                                       cpu_x->size[0U], cpu_a->size[1U],
                                       cpu_convOut->size[0U]);
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_cpu_convOut, 2, &sd_emlrtRTEI, true);
    c_cpu_convOut->size[0] = cpu_x->size[0];
    c_cpu_convOut->size[1] = cpu_convOut->size[1];
    nb = cpu_convOut->size[1] - 1;
    na = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    numIters = computeNumIters(nb, na);
    mwGetLaunchParameters1D(numIters, &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(c_cpu_convOut, &b_gpu_convOut, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel50#" MW_AT_LINE);
      filter_kernel50<<<grid, block>>>(gpu_convOut, na, nb, b_gpu_convOut,
                                       c_cpu_convOut->size[0U],
                                       cpu_convOut->size[0U]);
    }
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_convOut);
    zi[0] = cpu_x->size[0];
    zi[1] = cpu_x->size[1];
    na = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = cpu_x->size[0];
    cpu_y->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, na, &vd_emlrtRTEI);
    b_zi = zi[0] * zi[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(b_zi), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_y, gpu_y, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel51#" MW_AT_LINE);
      filter_kernel51<<<grid, block>>>(b_gpu_convOut, b_zi, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_cpu_convOut);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_kernel1D);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_kernel1D);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_convOut);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_convOut);
  *b_outdatedOnGpu = false;
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
  nvtxRangePop();
}

//
//
void filter(emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
            emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
            emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
            emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
            emxArray_real_T *cpu_x, emxArray_real_T *gpu_x,
            boolean_T *x_outdatedOnGpu, emxArray_real_T *cpu_zi,
            boolean_T *zi_outdatedOnCpu, emxArray_real_T *gpu_zi,
            boolean_T *zi_outdatedOnGpu, emxArray_real_T *cpu_y,
            boolean_T *y_outdatedOnCpu, emxArray_real_T *gpu_y,
            boolean_T *y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T gpu_convOut;
  emxArray_real_T gpu_input;
  emxArray_real_T gpu_kernel1D;
  emxArray_real_T *cpu_convOut;
  emxArray_real_T *cpu_input;
  emxArray_real_T *cpu_kernel1D;
  real_T *gpu_a1;
  int32_T na;
  int32_T nb;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#filter#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_convOut);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_kernel1D);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a1, 8UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[1];
  nb = cpu_b->size[1];
  nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
  }
  nvtxMarkA("#filter_kernel1#" MW_AT_LINE);
  filter_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
  cpuEmxAllocateOrResize_real_T(cpu_a, !*a_outdatedOnCpu);
  if (*a_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0])) &&
      (!(cpu_a->data[0] == 0.0)) && (cpu_a->data[0] != 1.0)) {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel2#" MW_AT_LINE);
      filter_kernel2<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel3#" MW_AT_LINE);
      filter_kernel3<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
    }
    nvtxMarkA("#filter_kernel4#" MW_AT_LINE);
    filter_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
    *a_outdatedOnCpu = true;
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_kernel1D, 1, &sd_emlrtRTEI, true);
  if (cpu_a->size[1] > cpu_b->size[1]) {
    cpu_kernel1D->size[0] = cpu_a->size[1];
    na = cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel5#" MW_AT_LINE);
      filter_kernel5<<<grid, block>>>(*gpu_b, na - 1, gpu_kernel1D);
    }
    na = cpu_a->size[1] - cpu_b->size[1];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel6#" MW_AT_LINE);
      filter_kernel6<<<grid, block>>>(na - 1, gpu_kernel1D, cpu_b->size[1U]);
    }
  } else {
    cpu_kernel1D->size[0] = cpu_b->size[1];
    na = cpu_b->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_kernel1D, &gpu_kernel1D, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel10#" MW_AT_LINE);
      filter_kernel10<<<grid, block>>>(*gpu_b, na, gpu_kernel1D);
    }
  }
  if (cpu_x->size[0] == 0) {
    *y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
  } else {
    dim3 b_grid;
    uint32_T varargin_3_idx_0;
    boolean_T convOut_outdatedOnCpu;
    boolean_T convOut_outdatedOnGpu;
    varargin_3_idx_0 = static_cast<uint32_T>(cpu_kernel1D->size[0]);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_input, 1, &td_emlrtRTEI, true);
    cpu_input->size[0] = cpu_x->size[0];
    na = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_input, &gpu_input, true);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_x, cpu_x);
    }
    *x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel7#" MW_AT_LINE);
      filter_kernel7<<<grid, block>>>(*gpu_x, na, gpu_input);
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_convOut, 1, &sd_emlrtRTEI, true);
    na = cpu_convOut->size[0];
    cpu_convOut->size[0] = (cpu_x->size[0] + cpu_kernel1D->size[0]) - 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_convOut, na, &ud_emlrtRTEI);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    b_grid =
        dim3((static_cast<uint32_T>(cpu_convOut->size[0]) >> 5) +
                 ((static_cast<uint32_T>(cpu_convOut->size[0]) & 31U) != 0U),
             1U, 1U);
    validLaunchParams = mwValidateLaunchParameters(b_grid, dim3(32U, 1U, 1U));
    if (validLaunchParams) {
      nvtxMarkA("#stencilfunKernel#" MW_AT_LINE);
      stencilfunKernel<<<b_grid, dim3(32U, 1U, 1U)>>>(
          gpu_input, static_cast<int32_T>(varargin_3_idx_0) - 1, 0.0,
          gpu_kernel1D, gpu_convOut, cpu_convOut->size[0U],
          cpu_kernel1D->size[0U], cpu_input->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    convOut_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_input);
    if (cpu_zi->size[0] != 0) {
      if (cpu_zi->size[0] == 1) {
        nvtxMarkA("#cpuEmxAllocateOrResize_real_T#" MW_AT_LINE);
        cpuEmxAllocateOrResize_real_T(cpu_zi, !*zi_outdatedOnCpu);
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        cpu_convOut->data[0] += cpu_zi->data[0];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      } else {
        na = cpu_zi->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#filter_kernel9#" MW_AT_LINE);
          filter_kernel9<<<grid, block>>>(*gpu_zi, na - 1, gpu_convOut);
        }
      }
    }
    na = cpu_x->size[0] + cpu_a->size[1];
    profileLoopStart("filter_loop_0", __LINE__, (na - 2) + 1, "");
    for (int32_T i{0}; i <= na - 2; i++) {
      nb = static_cast<int32_T>(std::fmin(static_cast<real_T>(i) + 1.0,
                                          static_cast<real_T>(cpu_a->size[1])));
      profileLoopStart("filter_loop_1", __LINE__, (nb - 2) + 1, "");
      for (int32_T j{0}; j <= nb - 2; j++) {
        if (convOut_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_convOut, &gpu_convOut);
        }
        if (*a_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
        }
        *a_outdatedOnCpu = false;
        cpu_convOut->data[i] -=
            cpu_convOut->data[(i - j) - 1] * cpu_a->data[j + 1];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    na = cpu_y->size[0];
    cpu_y->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, na, &vd_emlrtRTEI);
    na = cpu_x->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                            2147483647U);
    if (b_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_convOut, &gpu_convOut,
                                  !convOut_outdatedOnGpu);
    }
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_y, gpu_y, true);
    if (convOut_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#filter_kernel8#" MW_AT_LINE);
      filter_kernel8<<<grid, block>>>(gpu_convOut, na - 1, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_convOut);
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_kernel1D);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_kernel1D);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_input);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_convOut);
  *b_outdatedOnGpu = false;
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace coder

// End of code generation (filter.cu)

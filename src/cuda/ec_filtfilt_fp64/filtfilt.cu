#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filtfilt.cu
//
// Code generation for function 'filtfilt'
//

// Include files
#include "filtfilt.h"
#include "bsxfun.h"
#include "fillIn.h"
#include "filter.h"
#include "filtfilt_data.h"
#include "filtfilt_emxutil.h"
#include "filtfilt_mexutil.h"
#include "filtfilt_types.h"
#include "introsort.h"
#include "repmat.h"
#include "rt_nonfinite.h"
#include "sparse.h"
#include "warning.h"
#include "MWCUBLASUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "cs.h"
#include "makeCXSparseMatrix.h"
#include "nvtx3/nvToolsExt.h"
#include "solve_from_lu.h"
#include "solve_from_qr.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtMCInfo d_emlrtMCI{
    53,        // lineNo
    19,        // colNo
    "flt2str", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/lib/+coder/+internal/"
    "flt2str.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    123,                                                        // lineNo
    10,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    166,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    353,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    420,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    179,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo
    f_emlrtRTEI{
        76,                  // lineNo
        13,                  // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo g_emlrtRTEI{
    184,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    150,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    179,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    423,                                                        // lineNo
    36,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    356,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    304,                                                        // lineNo
    16,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    424,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    357,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    424,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    357,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    307,                                                        // lineNo
    20,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    424,                                                        // lineNo
    40,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    181,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    307,                                                        // lineNo
    28,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    425,                                                        // lineNo
    23,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    425,                                                        // lineNo
    31,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    358,                                                        // lineNo
    23,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    308,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    358,                                                        // lineNo
    31,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    308,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo bb_emlrtRTEI{
    358,                                                        // lineNo
    39,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo cb_emlrtRTEI{
    309,                                                        // lineNo
    20,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo db_emlrtRTEI{
    426,                                                        // lineNo
    38,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo eb_emlrtRTEI{
    309,                                                        // lineNo
    28,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fb_emlrtRTEI{
    427,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo gb_emlrtRTEI{
    427,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo hb_emlrtRTEI{
    309,                                                        // lineNo
    36,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ib_emlrtRTEI{
    359,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo jb_emlrtRTEI{
    429,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo kb_emlrtRTEI{
    360,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo lb_emlrtRTEI{
    429,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo mb_emlrtRTEI{
    360,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo nb_emlrtRTEI{
    429,                                                        // lineNo
    37,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ob_emlrtRTEI{
    429,                                                        // lineNo
    53,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo pb_emlrtRTEI{
    373,                                                        // lineNo
    21,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo qb_emlrtRTEI{
    430,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo rb_emlrtRTEI{
    373,                                                        // lineNo
    29,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo sb_emlrtRTEI{
    430,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo tb_emlrtRTEI{
    430,                                                        // lineNo
    34,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ub_emlrtRTEI{
    374,                                                        // lineNo
    18,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo vb_emlrtRTEI{
    432,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo wb_emlrtRTEI{
    374,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo xb_emlrtRTEI{
    390,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo yb_emlrtRTEI{
    1,                                                          // lineNo
    14,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ac_emlrtRTEI{
    300,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo bc_emlrtRTEI{
    416,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo cc_emlrtRTEI{
    423,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo dc_emlrtRTEI{
    304,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ec_emlrtRTEI{
    427,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fc_emlrtRTEI{
    360,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo gc_emlrtRTEI{
    430,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo hc_emlrtRTEI{
    374,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ic_emlrtRTEI{
    426,                                                        // lineNo
    10,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo kc_emlrtRTEI{
    713,                                                        // lineNo
    40,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo lc_emlrtRTEI{
    223,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo mc_emlrtRTEI{
    224,                                                        // lineNo
    1,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo nc_emlrtRTEI{
    716,                                                        // lineNo
    40,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo oc_emlrtRTEI{
    54,                                                               // lineNo
    9,                                                                // colNo
    "div",                                                            // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/div.m" // pName
};

static emlrtRTEInfo pc_emlrtRTEI{
    231,                                                        // lineNo
    25,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo qc_emlrtRTEI{
    232,                                                        // lineNo
    26,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    245,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo sc_emlrtRTEI{
    246,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    249,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    250,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    1647,     // lineNo
    27,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo wc_emlrtRTEI{
    125,      // lineNo
    44,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo xc_emlrtRTEI{
    1676,     // lineNo
    5,        // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo yc_emlrtRTEI{
    13,                                                                // lineNo
    1,                                                                 // colNo
    "sparse",                                                          // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/sparfun/sparse.m" // pName
};

static emlrtRTEInfo ad_emlrtRTEI{
    261,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo
    bd_emlrtRTEI{
        388,           // lineNo
        38,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo
    cd_emlrtRTEI{
        405,           // lineNo
        46,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo
    dd_emlrtRTEI{
        399,           // lineNo
        46,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo ed_emlrtRTEI{
    232,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    261,                                                        // lineNo
    19,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo gd_emlrtRTEI{
    119,      // lineNo
    13,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo hd_emlrtRTEI{
    120,      // lineNo
    13,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo id_emlrtRTEI{
    125,      // lineNo
    13,       // colNo
    "sparse", // fName
    "/usr/local/MATLAB/R2024b/toolbox/shared/coder/coder/+coder/+internal/"
    "@sparse/sparse.m" // pName
};

static emlrtRTEInfo
    jd_emlrtRTEI{
        457,           // lineNo
        63,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo
    kd_emlrtRTEI{
        394,           // lineNo
        25,            // colNo
        "CXSparseAPI", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
        "CXSparseAPI.m" // pName
    };

static emlrtRTEInfo ld_emlrtRTEI{
    30,                    // lineNo
    21,                    // colNo
    "applyScalarFunction", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/"
    "applyScalarFunction.m" // pName
};

static emlrtRTEInfo md_emlrtRTEI{
    733,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo nd_emlrtRTEI{
    738,                                                        // lineNo
    20,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo od_emlrtRTEI{
    731,                                                        // lineNo
    15,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ae_emlrtRTEI{
    505,                                                        // lineNo
    12,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo be_emlrtRTEI{
    661,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ce_emlrtRTEI{
    672,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo de_emlrtRTEI{
    667,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ee_emlrtRTEI{
    656,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo fe_emlrtRTEI{
    550,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ge_emlrtRTEI{
    673,                                                        // lineNo
    9,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo he_emlrtRTEI{
    551,                                                        // lineNo
    13,                                                         // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo ie_emlrtRTEI{
    657,                                                        // lineNo
    5,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

static emlrtRTEInfo je_emlrtRTEI{
    452,                                                        // lineNo
    8,                                                          // colNo
    "filtfilt",                                                 // fName
    "/usr/local/MATLAB/R2024b/toolbox/signal/signal/filtfilt.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[14]);

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location);

static void binary_expand_op_1(emxArray_real_T *in1, real_T in2, int32_T in3,
                               const emxArray_real_T *in4, int32_T in5,
                               const emxArray_real_T *in6);

namespace coder {
static void filtfiltParser(const emxArray_real_T *ctf,
                           const emxArray_real_T *varargin_1,
                           emxArray_real_T *B, emxArray_real_T *A,
                           real_T *numStage, real_T *P, real_T *Q,
                           boolean_T *isNumNonempty, boolean_T *isDenNonempty);

static void findEffectiveFilterLen(emxArray_real_T *coef, real_T effLen_data[],
                                   int32_T effLen_size[1]);

static void getCoeffsAndInitialConditions(const emxArray_real_T *num,
                                          const emxArray_real_T *den,
                                          real_T numStage, real_T P, real_T Q,
                                          emxArray_real_T *B,
                                          emxArray_real_T *A,
                                          emxArray_real_T *zi, real_T *Letr);

} // namespace coder
static int32_T div_s32(int32_T numerator, int32_T denominator);

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[14]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[14]);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[14])
{
  static const int32_T dims[2]{1, 14};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 14);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  const mxArray *m3;
  nvtxRangePushA("#fcn#b_sprintf#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  m3 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                             "sprintf", true, location);
  nvtxRangePop();
  return m3;
}

static void binary_expand_op_1(emxArray_real_T *in1, real_T in2, int32_T in3,
                               const emxArray_real_T *in4, int32_T in5,
                               const emxArray_real_T *in6)
{
  emxArray_real_T *b_in4;
  int32_T i;
  int32_T in6_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#binary_expand_op_1#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  in6_idx_0 = in6->size[0];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_in4, 2, &oc_emlrtRTEI, true);
  i = b_in4->size[0] * b_in4->size[1];
  if (in6_idx_0 == 1) {
    b_in4->size[0] = in4->size[0];
  } else {
    b_in4->size[0] = in6_idx_0;
  }
  b_in4->size[1] = in5 + 1;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_in4, i, &oc_emlrtRTEI);
  stride_0_0 = (in4->size[0] != 1);
  stride_1_0 = (in6_idx_0 != 1);
  profileLoopStart("binary_expand_op_1_loop_0", __LINE__, in5 + 1, "");
  for (i = 0; i <= in5; i++) {
    int32_T b;
    if (in6_idx_0 == 1) {
      b = in4->size[0];
    } else {
      b = in6_idx_0;
    }
    profileLoopStart("binary_expand_op_1_loop_2", __LINE__, (b - 1) + 1, "");
    for (int32_T i1{0}; i1 < b; i1++) {
      b_in4->data[i1 + b_in4->size[0] * i] =
          in4->data[i1 * stride_0_0 + in4->size[0] * i] /
          in6->data[i1 * stride_1_0];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  in6_idx_0 = static_cast<int32_T>(in2);
  profileLoopStart("binary_expand_op_1_loop_1", __LINE__, (in3 - 1) + 1, "");
  for (i = 0; i < in3; i++) {
    profileLoopStart("binary_expand_op_1_loop_3", __LINE__, (in6_idx_0 - 1) + 1,
                     "");
    for (int32_T i1{0}; i1 < in6_idx_0; i1++) {
      in1->data[i1 + in1->size[0] * i] =
          b_in4->data[i1 + static_cast<int32_T>(in2) * i];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_in4);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

//
//
namespace coder {
static void filtfiltParser(const emxArray_real_T *ctf,
                           const emxArray_real_T *varargin_1,
                           emxArray_real_T *B, emxArray_real_T *A,
                           real_T *numStage, real_T *P, real_T *Q,
                           boolean_T *isNumNonempty, boolean_T *isDenNonempty)
{
  emxArray_real_T *NUM;
  int32_T b_numStage;
  int32_T numSV;
  int32_T u0;
  int32_T u1;
  boolean_T guard1;
  boolean_T guard2;
  boolean_T isDenNonemptyVector;
  boolean_T isDenVector;
  boolean_T isNumVector;
  nvtxRangePushA("#fcn#filtfiltParser#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  *isNumNonempty = ((ctf->size[0] != 0) && (ctf->size[1] != 0));
  *isDenNonempty = ((varargin_1->size[0] != 0) && (varargin_1->size[1] != 0));
  isNumVector = ((ctf->size[0] == 1) || (ctf->size[1] == 1));
  isDenVector = ((varargin_1->size[0] == 1) || (varargin_1->size[1] == 1));
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&NUM, 2, &je_emlrtRTEI, true);
  u0 = NUM->size[0] * NUM->size[1];
  NUM->size[0] = ctf->size[0];
  NUM->size[1] = ctf->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(NUM, u0, &ae_emlrtRTEI);
  profileLoopStart("filtfiltParser_loop_0", __LINE__,
                   (ctf->size[0] * ctf->size[1] - 1) + 1, "");
  for (u0 = 0; u0 < ctf->size[0] * ctf->size[1]; u0++) {
    NUM->data[u0] = ctf->data[u0];
  }
  profileLoopEnd();
  b_numStage = ctf->size[0];
  if ((*isDenNonempty) && isDenVector) {
    isDenNonemptyVector = true;
  } else {
    isDenNonemptyVector = false;
  }
  if ((ctf->size[1] == 6) && (ctf->size[0] == 1) && isDenNonemptyVector) {
    if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
      numSV = 0;
    } else {
      u0 = varargin_1->size[0];
      numSV = varargin_1->size[1];
      if (u0 >= numSV) {
        numSV = u0;
      }
    }
    if (numSV <= 2) {
      if (ctf->data[3] == 1.0) {
        nvtxMarkA("#warning#" MW_AT_LINE);
        internal::warning();
      } else {
        nvtxMarkA("#b_warning#" MW_AT_LINE);
        internal::b_warning();
      }
    }
  }
  guard1 = false;
  guard2 = false;
  if ((*isNumNonempty) && isDenNonemptyVector && (ctf->size[1] == 6)) {
    if (ctf->size[0] > 1) {
      guard2 = true;
    } else if (ctf->data[3] == 1.0) {
      if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
        numSV = 0;
      } else {
        u0 = varargin_1->size[0];
        numSV = varargin_1->size[1];
        if (u0 >= numSV) {
          numSV = u0;
        }
      }
      if (numSV <= 2) {
        guard2 = true;
      } else {
        guard1 = true;
      }
    } else {
      guard1 = true;
    }
  } else {
    guard1 = true;
  }
  if (guard2) {
    if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
      numSV = 0;
    } else {
      u0 = varargin_1->size[0];
      numSV = varargin_1->size[1];
      if (u0 >= numSV) {
        numSV = u0;
      }
    }
    if (static_cast<uint32_T>(numSV) ==
        static_cast<uint32_T>(ctf->size[0]) + 1U) {
      NUM->data[ctf->size[0] - 1] =
          varargin_1->data[ctf->size[0]] * ctf->data[ctf->size[0] - 1];
      NUM->data[(ctf->size[0] + NUM->size[0]) - 1] =
          varargin_1->data[ctf->size[0]] *
          ctf->data[(ctf->size[0] + ctf->size[0]) - 1];
      NUM->data[(ctf->size[0] + NUM->size[0] * 2) - 1] =
          varargin_1->data[ctf->size[0]] *
          ctf->data[(ctf->size[0] + ctf->size[0] * 2) - 1];
      numSV--;
    }
    profileLoopStart("filtfiltParser_loop_1", __LINE__, (numSV - 1) + 1, "");
    for (u0 = 0; u0 < numSV; u0++) {
      NUM->data[u0] *= varargin_1->data[u0];
      NUM->data[u0 + NUM->size[0]] *= varargin_1->data[u0];
      NUM->data[u0 + NUM->size[0] * 2] *= varargin_1->data[u0];
    }
    profileLoopEnd();
    u0 = B->size[0] * B->size[1];
    B->size[0] = NUM->size[0];
    B->size[1] = 3;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(B, u0, &fe_emlrtRTEI);
    u0 = A->size[0] * A->size[1];
    A->size[0] = NUM->size[0];
    A->size[1] = 3;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(A, u0, &he_emlrtRTEI);
    profileLoopStart("filtfiltParser_loop_7", __LINE__, 2 + 1, "");
    for (u0 = 0; u0 < 3; u0++) {
      profileLoopStart("filtfiltParser_loop_8", __LINE__,
                       (NUM->size[0] - 1) + 1, "");
      for (numSV = 0; numSV < NUM->size[0]; numSV++) {
        B->data[numSV + B->size[0] * u0] = NUM->data[numSV + NUM->size[0] * u0];
        A->data[numSV + A->size[0] * u0] =
            NUM->data[numSV + NUM->size[0] * (u0 + 3)];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    numSV = 3;
    u1 = 3;
  }
  if (guard1) {
    if (isNumVector && isDenVector) {
      if ((ctf->size[0] == 0) || (ctf->size[1] == 0)) {
        numSV = 0;
      } else {
        u0 = ctf->size[0];
        numSV = ctf->size[1];
        if (u0 >= numSV) {
          numSV = u0;
        }
      }
      if ((varargin_1->size[0] == 0) || (varargin_1->size[1] == 0)) {
        u1 = 0;
      } else {
        u0 = varargin_1->size[0];
        u1 = varargin_1->size[1];
        if (u0 >= u1) {
          u1 = u0;
        }
      }
      b_numStage = 1;
      u0 = B->size[0] * B->size[1];
      B->size[0] = 1;
      B->size[1] = numSV;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(B, u0, &ee_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_5", __LINE__, (numSV - 1) + 1, "");
      for (u0 = 0; u0 < numSV; u0++) {
        B->data[u0] = ctf->data[u0];
      }
      profileLoopEnd();
      u0 = A->size[0] * A->size[1];
      A->size[0] = 1;
      A->size[1] = u1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(A, u0, &ie_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_9", __LINE__, (u1 - 1) + 1, "");
      for (u0 = 0; u0 < u1; u0++) {
        A->data[u0] = varargin_1->data[u0];
      }
      profileLoopEnd();
    } else if ((*isNumNonempty) && (!isNumVector) &&
               ((varargin_1->size[0] == 1) && (varargin_1->size[1] == 1))) {
      u0 = B->size[0] * B->size[1];
      B->size[0] = ctf->size[0];
      B->size[1] = ctf->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(B, u0, &be_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_2", __LINE__,
                       (ctf->size[0] * ctf->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < ctf->size[0] * ctf->size[1]; u0++) {
        B->data[u0] = ctf->data[u0];
      }
      profileLoopEnd();
      numSV = ctf->size[1];
      b_numStage = ctf->size[0];
      u1 = 1;
      nvtxMarkA("#repmat#" MW_AT_LINE);
      repmat(varargin_1, static_cast<real_T>(ctf->size[0]), A);
    } else if ((*isDenNonempty) && (!isDenVector) &&
               ((ctf->size[0] == 1) && (ctf->size[1] == 1))) {
      u0 = A->size[0] * A->size[1];
      A->size[0] = varargin_1->size[0];
      A->size[1] = varargin_1->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(A, u0, &de_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_4", __LINE__,
                       (varargin_1->size[0] * varargin_1->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < varargin_1->size[0] * varargin_1->size[1]; u0++) {
        A->data[u0] = varargin_1->data[u0];
      }
      profileLoopEnd();
      u1 = varargin_1->size[1];
      b_numStage = varargin_1->size[0];
      numSV = 1;
      nvtxMarkA("#repmat#" MW_AT_LINE);
      repmat(ctf, static_cast<real_T>(varargin_1->size[0]), B);
    } else {
      u0 = B->size[0] * B->size[1];
      B->size[0] = ctf->size[0];
      B->size[1] = ctf->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(B, u0, &ce_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_3", __LINE__,
                       (ctf->size[0] * ctf->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < ctf->size[0] * ctf->size[1]; u0++) {
        B->data[u0] = ctf->data[u0];
      }
      profileLoopEnd();
      u0 = A->size[0] * A->size[1];
      A->size[0] = varargin_1->size[0];
      A->size[1] = varargin_1->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(A, u0, &ge_emlrtRTEI);
      profileLoopStart("filtfiltParser_loop_6", __LINE__,
                       (varargin_1->size[0] * varargin_1->size[1] - 1) + 1, "");
      for (u0 = 0; u0 < varargin_1->size[0] * varargin_1->size[1]; u0++) {
        A->data[u0] = varargin_1->data[u0];
      }
      profileLoopEnd();
      numSV = ctf->size[1];
      b_numStage = ctf->size[0];
      u1 = varargin_1->size[1];
    }
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&NUM);
  *numStage = b_numStage;
  *P = numSV;
  *Q = u1;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

//
//
static void findEffectiveFilterLen(emxArray_real_T *coef, real_T effLen_data[],
                                   int32_T effLen_size[1])
{
  emxArray_boolean_T *x;
  emxArray_real_T *varargin_1;
  real_T maxCoef;
  int32_T ii_data[1];
  int32_T idx;
  int32_T k;
  int32_T nx;
  boolean_T exitg1;
  nvtxRangePushA("#fcn#findEffectiveFilterLen#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nx = coef->size[1];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&varargin_1, 2, &od_emlrtRTEI, true);
  idx = varargin_1->size[0] * varargin_1->size[1];
  varargin_1->size[0] = 1;
  varargin_1->size[1] = coef->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(varargin_1, idx, &ld_emlrtRTEI);
  profileLoopStart("findEffectiveFilterLen_loop_0", __LINE__, (nx - 1) + 1, "");
  for (k = 0; k < nx; k++) {
    varargin_1->data[k] = std::abs(coef->data[k]);
  }
  profileLoopEnd();
  nx = varargin_1->size[1];
  maxCoef = varargin_1->data[0];
  profileLoopStart("findEffectiveFilterLen_loop_1", __LINE__, (nx - 2) + 1, "");
  for (idx = 0; idx <= nx - 2; idx++) {
    real_T d;
    boolean_T p;
    d = varargin_1->data[idx + 1];
    if (std::isnan(d)) {
      p = false;
    } else if (std::isnan(maxCoef)) {
      p = true;
    } else {
      p = (maxCoef < d);
    }
    if (p) {
      maxCoef = d;
    }
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&varargin_1);
  if (maxCoef != 0.0) {
    idx = coef->size[1];
    nx = coef->size[0] * coef->size[1];
    coef->size[0] = 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(coef, nx, &md_emlrtRTEI);
    profileLoopStart("findEffectiveFilterLen_loop_2", __LINE__, (idx - 1) + 1,
                     "");
    for (nx = 0; nx < idx; nx++) {
      coef->data[nx] /= maxCoef;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxInit_boolean_T#" MW_AT_LINE);
  emxInit_boolean_T(&x, 1, &nd_emlrtRTEI, true);
  idx = x->size[0];
  x->size[0] = coef->size[1];
  nvtxMarkA("#emxEnsureCapacity_boolean_T#" MW_AT_LINE);
  emxEnsureCapacity_boolean_T(x, idx, &nd_emlrtRTEI);
  profileLoopStart("findEffectiveFilterLen_loop_3", __LINE__,
                   (coef->size[1] - 1) + 1, "");
  for (idx = 0; idx < coef->size[1]; idx++) {
    x->data[idx] = (coef->data[idx] != 0.0);
  }
  profileLoopEnd();
  k = (x->size[0] >= 1);
  nx = x->size[0];
  idx = 0;
  exitg1 = false;
  nvtxRangePushA("#loop#findEffectiveFilterLen_whileloop_0##" MW_AT_LINE);
  while ((!exitg1) && (nx > 0)) {
    if (x->data[nx - 1]) {
      idx = 1;
      ii_data[0] = nx;
      exitg1 = true;
    } else {
      nx--;
    }
  }
  nvtxRangePop();
  nvtxMarkA("#emxFree_boolean_T#" MW_AT_LINE);
  emxFree_boolean_T(&x);
  if (k == 1) {
    if (idx == 0) {
      k = 0;
    }
  } else {
    k = (idx >= 1);
  }
  effLen_size[0] = k;
  profileLoopStart("findEffectiveFilterLen_loop_4", __LINE__, (k - 1) + 1, "");
  for (idx = 0; idx < k; idx++) {
    effLen_data[0] = ii_data[0];
  }
  profileLoopEnd();
  if (k == 0) {
    effLen_size[0] = 1;
    effLen_data[0] = 0.0;
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

//
//
static void getCoeffsAndInitialConditions(const emxArray_real_T *num,
                                          const emxArray_real_T *den,
                                          real_T numStage, real_T P, real_T Q,
                                          emxArray_real_T *B,
                                          emxArray_real_T *A,
                                          emxArray_real_T *zi, real_T *Letr)
{
  static const int32_T iv[2]{1, 6};
  static const char_T rfmt[6]{'%', '1', '4', '.', '6', 'e'};
  emxArray_int32_T *b_in_colidx;
  emxArray_int32_T *b_in_rowidx;
  emxArray_int32_T *b_t;
  emxArray_int32_T *c_in_colidx;
  emxArray_int32_T *c_in_rowidx;
  emxArray_int32_T *cidxInt;
  emxArray_int32_T *in_colidx;
  emxArray_int32_T *in_rowidx;
  emxArray_int32_T *ridxInt;
  emxArray_int32_T *sortedIndices;
  emxArray_int32_T *t;
  emxArray_int32_T *y_colidx;
  emxArray_int32_T *y_rowidx;
  emxArray_real_T *b_den;
  emxArray_real_T *b_in_d;
  emxArray_real_T *b_num;
  emxArray_real_T *b_outBuff;
  emxArray_real_T *c_den;
  emxArray_real_T *c_in_d;
  emxArray_real_T *c_num;
  emxArray_real_T *colIdx;
  emxArray_real_T *in_d;
  emxArray_real_T *outBuff;
  emxArray_real_T *rhs;
  emxArray_real_T *rowIdx;
  emxArray_real_T *vals;
  emxArray_real_T *y_d;
  emxArray_real_T *zik;
  const mxArray *b_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *y;
  real_T b_tmp_data[1];
  real_T tmp_data[1];
  real_T M;
  real_T ord;
  int32_T b_loop_ub;
  int32_T b_ns;
  int32_T c_ns;
  int32_T cptr;
  int32_T i;
  int32_T i10;
  int32_T i11;
  int32_T loop_ub;
  int32_T nc;
  int32_T numalloc{0};
  int32_T ny;
  int32_T thism;
  int32_T vlen;
  nvtxRangePushA("#fcn#getCoeffsAndInitialConditions#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  M = std::fmax(P, Q);
  ord = 0.0;
  ny = static_cast<int32_T>(numStage);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_num, 2, &kc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_den, 2, &nc_emlrtRTEI, true);
  profileLoopStart("getCoeffsAndInitialConditions_loop_0", __LINE__,
                   (ny - 1) + 1, "");
  for (int32_T ns{0}; ns < ny; ns++) {
    i = b_num->size[0] * b_num->size[1];
    b_num->size[0] = 1;
    b_num->size[1] = num->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b_num, i, &kc_emlrtRTEI);
    profileLoopStart("getCoeffsAndInitialConditions_loop_1", __LINE__,
                     (num->size[1] - 1) + 1, "");
    for (i = 0; i < num->size[1]; i++) {
      b_num->data[i] = num->data[ns + num->size[0] * i];
    }
    profileLoopEnd();
    i = b_den->size[0] * b_den->size[1];
    b_den->size[0] = 1;
    b_den->size[1] = den->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b_den, i, &nc_emlrtRTEI);
    profileLoopStart("getCoeffsAndInitialConditions_loop_3", __LINE__,
                     (den->size[1] - 1) + 1, "");
    for (i = 0; i < den->size[1]; i++) {
      b_den->data[i] = den->data[ns + den->size[0] * i];
    }
    int32_T tmp_size[1];
    profileLoopEnd();
    nvtxMarkA("#findEffectiveFilterLen#" MW_AT_LINE);
    findEffectiveFilterLen(b_num, tmp_data, tmp_size);
    nvtxMarkA("#findEffectiveFilterLen#" MW_AT_LINE);
    findEffectiveFilterLen(b_den, b_tmp_data, tmp_size);
    ord = (ord + std::fmax(tmp_data[0], b_tmp_data[0])) - 1.0;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_den);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_num);
  i = B->size[0] * B->size[1];
  B->size[0] = static_cast<int32_T>(numStage);
  B->size[1] = static_cast<int32_T>(M);
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(B, i, &lc_emlrtRTEI);
  i = A->size[0] * A->size[1];
  A->size[0] = static_cast<int32_T>(numStage);
  A->size[1] = static_cast<int32_T>(M);
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(A, i, &mc_emlrtRTEI);
  profileLoopStart(
      "getCoeffsAndInitialConditions_loop_2", __LINE__,
      (static_cast<int32_T>(numStage) * static_cast<int32_T>(M) - 1) + 1, "");
  for (i = 0; i < static_cast<int32_T>(numStage) * static_cast<int32_T>(M);
       i++) {
    B->data[i] = 0.0;
    A->data[i] = 0.0;
  }
  profileLoopEnd();
  if (P < 1.0) {
    cptr = 0;
    ny = 0;
  } else {
    cptr = static_cast<int32_T>(P);
    ny = static_cast<int32_T>(P);
  }
  if (num->size[0] == den->size[0]) {
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_num, 2, &oc_emlrtRTEI, true);
    i = c_num->size[0] * c_num->size[1];
    c_num->size[0] = num->size[0];
    c_num->size[1] = cptr;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(c_num, i, &oc_emlrtRTEI);
    profileLoopStart("getCoeffsAndInitialConditions_loop_4", __LINE__,
                     (cptr - 1) + 1, "");
    for (i = 0; i < cptr; i++) {
      profileLoopStart("getCoeffsAndInitialConditions_loop_5", __LINE__,
                       (num->size[0] - 1) + 1, "");
      for (thism = 0; thism < num->size[0]; thism++) {
        c_num->data[thism + c_num->size[0] * i] =
            num->data[thism + num->size[0] * i] / den->data[thism];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    vlen = static_cast<int32_T>(numStage);
    profileLoopStart("getCoeffsAndInitialConditions_loop_6", __LINE__,
                     (ny - 1) + 1, "");
    for (i = 0; i < ny; i++) {
      profileLoopStart("getCoeffsAndInitialConditions_loop_8", __LINE__,
                       (vlen - 1) + 1, "");
      for (thism = 0; thism < vlen; thism++) {
        B->data[thism + B->size[0] * i] =
            c_num->data[thism + static_cast<int32_T>(numStage) * i];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_num);
  } else {
    nvtxMarkA("#binary_expand_op_1#" MW_AT_LINE);
    binary_expand_op_1(B, numStage, ny, num, cptr - 1, den);
  }
  if (Q < 1.0) {
    cptr = 0;
  } else {
    cptr = static_cast<int32_T>(Q);
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_den, 2, &oc_emlrtRTEI, true);
  i = c_den->size[0] * c_den->size[1];
  c_den->size[0] = den->size[0];
  c_den->size[1] = cptr;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(c_den, i, &oc_emlrtRTEI);
  profileLoopStart("getCoeffsAndInitialConditions_loop_7", __LINE__,
                   (cptr - 1) + 1, "");
  for (i = 0; i < cptr; i++) {
    profileLoopStart("getCoeffsAndInitialConditions_loop_9", __LINE__,
                     (den->size[0] - 1) + 1, "");
    for (thism = 0; thism < den->size[0]; thism++) {
      c_den->data[thism + c_den->size[0] * i] =
          den->data[thism + den->size[0] * i] / den->data[thism];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  vlen = static_cast<int32_T>(numStage);
  if (Q < 1.0) {
    cptr = 0;
  } else {
    cptr = static_cast<int32_T>(Q);
  }
  profileLoopStart("getCoeffsAndInitialConditions_loop_10", __LINE__,
                   (cptr - 1) + 1, "");
  for (i = 0; i < cptr; i++) {
    profileLoopStart("getCoeffsAndInitialConditions_loop_11", __LINE__,
                     (vlen - 1) + 1, "");
    for (thism = 0; thism < vlen; thism++) {
      A->data[thism + A->size[0] * i] =
          c_den->data[thism + static_cast<int32_T>(numStage) * i];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_den);
  if (M > 1.0) {
    int32_T i7;
    int32_T i9;
    i = zi->size[0] * zi->size[1];
    zi->size[0] = static_cast<int32_T>(M - 1.0);
    zi->size[1] = static_cast<int32_T>(numStage);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(zi, i, &pc_emlrtRTEI);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&rhs, 1, &ed_emlrtRTEI, true);
    i = rhs->size[0];
    rhs->size[0] = static_cast<int32_T>(M - 1.0);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(rhs, i, &qc_emlrtRTEI);
    if (static_cast<int32_T>((M - 1.0) - 1.0) + 1 < 2) {
      ny = 0;
      vlen = -1;
    } else {
      ny = 1;
      vlen = static_cast<int32_T>((M - 1.0) - 1.0);
    }
    if (static_cast<real_T>(static_cast<int32_T>(M - 1.0)) - 1.0 < 1.0) {
      i7 = 1;
    } else {
      i7 = static_cast<int32_T>(M - 1.0);
    }
    if (static_cast<int32_T>((M - 1.0) - 1.0) < 1) {
      cptr = 0;
    } else {
      cptr = static_cast<int32_T>((M - 1.0) - 1.0);
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&rowIdx, 2, &rc_emlrtRTEI, true);
    i = rowIdx->size[0] * rowIdx->size[1];
    rowIdx->size[0] = 1;
    rowIdx->size[1] =
        (((static_cast<int32_T>((M - 1.0) - 1.0) + vlen) - ny) + cptr) + 2;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(rowIdx, i, &rc_emlrtRTEI);
    thism = static_cast<int32_T>((M - 1.0) - 1.0);
    profileLoopStart("getCoeffsAndInitialConditions_loop_12", __LINE__,
                     thism + 1, "");
    for (i = 0; i <= thism; i++) {
      rowIdx->data[i] = static_cast<real_T>(i) + 1.0;
    }
    profileLoopEnd();
    thism = vlen - ny;
    profileLoopStart("getCoeffsAndInitialConditions_loop_13", __LINE__,
                     thism + 1, "");
    for (i = 0; i <= thism; i++) {
      rowIdx->data[(i + static_cast<int32_T>((M - 1.0) - 1.0)) + 1] =
          static_cast<real_T>(ny + i) + 1.0;
    }
    profileLoopEnd();
    profileLoopStart("getCoeffsAndInitialConditions_loop_14", __LINE__,
                     (cptr - 1) + 1, "");
    for (i = 0; i < cptr; i++) {
      rowIdx->data[(((i + static_cast<int32_T>((M - 1.0) - 1.0)) + vlen) - ny) +
                   2] = static_cast<real_T>(i) + 1.0;
    }
    profileLoopEnd();
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&colIdx, 2, &sc_emlrtRTEI, true);
    i = colIdx->size[0] * colIdx->size[1];
    colIdx->size[0] = 1;
    colIdx->size[1] =
        ((((static_cast<int32_T>(M - 1.0) + vlen) - ny) + vlen) - ny) + 2;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(colIdx, i, &sc_emlrtRTEI);
    thism = static_cast<int32_T>(M - 1.0);
    profileLoopStart("getCoeffsAndInitialConditions_loop_15", __LINE__,
                     (thism - 1) + 1, "");
    for (i = 0; i < thism; i++) {
      colIdx->data[i] = 1.0;
    }
    profileLoopEnd();
    thism = vlen - ny;
    profileLoopStart("getCoeffsAndInitialConditions_loop_16", __LINE__,
                     thism + 1, "");
    for (i = 0; i <= thism; i++) {
      colIdx->data[i + static_cast<int32_T>(M - 1.0)] =
          static_cast<real_T>(ny + i) + 1.0;
    }
    profileLoopEnd();
    thism = vlen - ny;
    profileLoopStart("getCoeffsAndInitialConditions_loop_17", __LINE__,
                     thism + 1, "");
    for (i = 0; i <= thism; i++) {
      colIdx->data[(((i + static_cast<int32_T>(M - 1.0)) + vlen) - ny) + 1] =
          static_cast<real_T>(ny + i) + 1.0;
    }
    profileLoopEnd();
    i9 = static_cast<int32_T>(numStage);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&y_d, 1, &fd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&y_colidx, 1, &fd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&y_rowidx, 1, &fd_emlrtRTEI, true);
    if (static_cast<int32_T>(numStage) - 1 >= 0) {
      if (M < 3.0) {
        i10 = 0;
        i11 = 0;
      } else {
        i10 = 2;
        i11 = static_cast<int32_T>(M);
      }
      loop_ub = i11 - i10;
      b_loop_ub = i7;
      nc = colIdx->size[1];
      b_ns = rowIdx->size[1];
      c_ns = colIdx->size[1];
      numalloc = colIdx->size[1];
      if (numalloc < 1) {
        numalloc = 1;
      }
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&vals, 2, &tc_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&ridxInt, 1, &gd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&cidxInt, 1, &hd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&sortedIndices, 1, &id_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&t, 1, &xc_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_t, 1, &xc_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&zik, 1, &ad_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&in_d, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&in_colidx, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&in_rowidx, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_in_d, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_in_colidx, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&b_in_rowidx, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c_in_d, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&c_in_colidx, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
    emxInit_int32_T(&c_in_rowidx, 1, &jd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&outBuff, 1, &kd_emlrtRTEI, true);
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&b_outBuff, 1, &kd_emlrtRTEI, true);
    profileLoopStart("getCoeffsAndInitialConditions_loop_18", __LINE__,
                     (i9 - 1) + 1, "");
    for (int32_T ns{0}; ns < i9; ns++) {
      i = vals->size[0] * vals->size[1];
      vals->size[0] = 1;
      vals->size[1] = (((i11 - i10) + i7) + i7) - 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(vals, i, &tc_emlrtRTEI);
      vals->data[0] = A->data[ns + A->size[0]] + 1.0;
      profileLoopStart("getCoeffsAndInitialConditions_loop_19", __LINE__,
                       (loop_ub - 1) + 1, "");
      for (i = 0; i < loop_ub; i++) {
        vals->data[i + 1] = A->data[ns + A->size[0] * (i10 + i)];
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_20", __LINE__,
                       (b_loop_ub - 2) + 1, "");
      for (i = 0; i <= b_loop_ub - 2; i++) {
        vals->data[((i + i11) - i10) + 1] = 1.0;
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_21", __LINE__,
                       (b_loop_ub - 2) + 1, "");
      for (i = 0; i <= b_loop_ub - 2; i++) {
        vals->data[((i + i11) - i10) + i7] = -1.0;
      }
      profileLoopEnd();
      M = B->data[ns];
      cptr = rhs->size[0];
      i = rhs->size[0];
      rhs->size[0] = cptr;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(rhs, i, &uc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_22", __LINE__,
                       (cptr - 1) + 1, "");
      for (i = 0; i < cptr; i++) {
        rhs->data[i] = B->data[ns + B->size[0] * (i + 1)] -
                       M * A->data[ns + A->size[0] * (i + 1)];
      }
      profileLoopEnd();
      i = ridxInt->size[0];
      ridxInt->size[0] = rowIdx->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(ridxInt, i, &vc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_23", __LINE__,
                       (b_ns - 1) + 1, "");
      for (vlen = 0; vlen < b_ns; vlen++) {
        ridxInt->data[vlen] = static_cast<int32_T>(rowIdx->data[vlen]);
      }
      profileLoopEnd();
      i = cidxInt->size[0];
      cidxInt->size[0] = colIdx->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cidxInt, i, &vc_emlrtRTEI);
      i = sortedIndices->size[0];
      sortedIndices->size[0] = colIdx->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(sortedIndices, i, &wc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_24", __LINE__,
                       (c_ns - 1) + 1, "");
      for (vlen = 0; vlen < c_ns; vlen++) {
        cidxInt->data[vlen] = static_cast<int32_T>(colIdx->data[vlen]);
        sortedIndices->data[vlen] = vlen + 1;
      }
      profileLoopEnd();
      nvtxMarkA("#introsort#" MW_AT_LINE);
      internal::introsort(sortedIndices, cidxInt->size[0], cidxInt, ridxInt);
      ny = cidxInt->size[0];
      i = t->size[0];
      t->size[0] = cidxInt->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(t, i, &xc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_25", __LINE__,
                       (cidxInt->size[0] - 1) + 1, "");
      for (i = 0; i < cidxInt->size[0]; i++) {
        t->data[i] = cidxInt->data[i];
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_26", __LINE__,
                       (ny - 1) + 1, "");
      for (vlen = 0; vlen < ny; vlen++) {
        cidxInt->data[vlen] = t->data[sortedIndices->data[vlen] - 1];
      }
      profileLoopEnd();
      ny = ridxInt->size[0];
      i = b_t->size[0];
      b_t->size[0] = ridxInt->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(b_t, i, &xc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_27", __LINE__,
                       (ridxInt->size[0] - 1) + 1, "");
      for (i = 0; i < ridxInt->size[0]; i++) {
        b_t->data[i] = ridxInt->data[i];
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_28", __LINE__,
                       (ny - 1) + 1, "");
      for (vlen = 0; vlen < ny; vlen++) {
        ridxInt->data[vlen] = b_t->data[sortedIndices->data[vlen] - 1];
      }
      profileLoopEnd();
      vlen = ridxInt->size[0];
      thism = ridxInt->data[0];
      profileLoopStart("getCoeffsAndInitialConditions_loop_29", __LINE__,
                       (vlen - 2) + 1, "");
      for (ny = 0; ny <= vlen - 2; ny++) {
        if (thism < ridxInt->data[ny + 1]) {
          thism = ridxInt->data[ny + 1];
        }
      }
      profileLoopEnd();
      ny = cidxInt->data[cidxInt->size[0] - 1];
      i = y_d->size[0];
      y_d->size[0] = numalloc;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(y_d, i, &yc_emlrtRTEI);
      if (numalloc - 1 >= 0) {
        std::memset(&y_d->data[0], 0,
                    static_cast<uint32_T>(numalloc) * sizeof(real_T));
      }
      i = y_colidx->size[0];
      y_colidx->size[0] = cidxInt->data[cidxInt->size[0] - 1] + 1;
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(y_colidx, i, &yc_emlrtRTEI);
      profileLoopStart("getCoeffsAndInitialConditions_loop_30", __LINE__,
                       cidxInt->data[cidxInt->size[0] - 1] + 1, "");
      for (i = 0; i <= cidxInt->data[cidxInt->size[0] - 1]; i++) {
        y_colidx->data[i] = 0;
      }
      profileLoopEnd();
      y_colidx->data[0] = 1;
      i = y_rowidx->size[0];
      y_rowidx->size[0] = numalloc;
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(y_rowidx, i, &yc_emlrtRTEI);
      if (numalloc - 1 >= 0) {
        std::memset(&y_rowidx->data[0], 0,
                    static_cast<uint32_T>(numalloc) * sizeof(int32_T));
      }
      cptr = 0;
      profileLoopStart("getCoeffsAndInitialConditions_loop_31", __LINE__,
                       (ny - 1) + 1, "");
      for (vlen = 0; vlen < ny; vlen++) {
        nvtxRangePushA(
            "#loop#getCoeffsAndInitialConditions_whileloop_0##" MW_AT_LINE);
        while ((cptr + 1 <= nc) && (cidxInt->data[cptr] == vlen + 1)) {
          y_rowidx->data[cptr] = ridxInt->data[cptr];
          cptr++;
        }
        nvtxRangePop();
        y_colidx->data[vlen + 1] = cptr + 1;
      }
      profileLoopEnd();
      profileLoopStart("getCoeffsAndInitialConditions_loop_32", __LINE__,
                       (nc - 1) + 1, "");
      for (vlen = 0; vlen < nc; vlen++) {
        y_d->data[vlen] = vals->data[sortedIndices->data[vlen] - 1];
      }
      profileLoopEnd();
      ny = cidxInt->data[cidxInt->size[0] - 1];
      nvtxMarkA("#sparse_fillIn#" MW_AT_LINE);
      sparse_fillIn(y_d, y_colidx, y_rowidx);
      if ((thism == 0) || (cidxInt->data[cidxInt->size[0] - 1] == 0)) {
        i = zik->size[0];
        zik->size[0] = cidxInt->data[cidxInt->size[0] - 1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(zik, i, &ad_emlrtRTEI);
        if (ny - 1 >= 0) {
          std::memset(&zik->data[0], 0,
                      static_cast<uint32_T>(ny) * sizeof(real_T));
        }
      } else if (rhs->size[0] == cidxInt->data[cidxInt->size[0] - 1]) {
        cs_di *b_cxA;
        cs_din *b_N;
        cs_dis *b_S;
        if (thism < cidxInt->data[cidxInt->size[0] - 1]) {
          nvtxMarkA("#sparse_ctranspose#" MW_AT_LINE);
          sparse_ctranspose(y_d, y_colidx, y_rowidx, thism,
                            cidxInt->data[cidxInt->size[0] - 1], b_in_d,
                            b_in_colidx, b_in_rowidx, &cptr, &ny);
          b_cxA = makeCXSparseMatrix(
              b_in_colidx->data[b_in_colidx->size[0] - 1] - 1, ny, cptr,
              &b_in_colidx->data[0], &b_in_rowidx->data[0], &b_in_d->data[0]);
        } else {
          b_cxA = makeCXSparseMatrix(y_colidx->data[y_colidx->size[0] - 1] - 1,
                                     cidxInt->data[cidxInt->size[0] - 1], thism,
                                     &y_colidx->data[0], &y_rowidx->data[0],
                                     &y_d->data[0]);
        }
        b_S = cs_di_sqr(2, b_cxA, 0);
        b_N = cs_di_lu(b_cxA, b_S, 1);
        cs_di_spfree(b_cxA);
        if (b_N == nullptr) {
          cs_di *c_cxA;
          cs_din *c_N;
          cs_dis *c_S;
          nvtxMarkA("#c_warning#" MW_AT_LINE);
          internal::c_warning();
          cs_di_sfree(b_S);
          cs_di_nfree(b_N);
          if (thism < cidxInt->data[cidxInt->size[0] - 1]) {
            nvtxMarkA("#sparse_ctranspose#" MW_AT_LINE);
            sparse_ctranspose(y_d, y_colidx, y_rowidx, thism,
                              cidxInt->data[cidxInt->size[0] - 1], c_in_d,
                              c_in_colidx, c_in_rowidx, &cptr, &ny);
            c_cxA = makeCXSparseMatrix(
                c_in_colidx->data[c_in_colidx->size[0] - 1] - 1, ny, cptr,
                &c_in_colidx->data[0], &c_in_rowidx->data[0], &c_in_d->data[0]);
          } else {
            c_cxA = makeCXSparseMatrix(
                y_colidx->data[y_colidx->size[0] - 1] - 1,
                cidxInt->data[cidxInt->size[0] - 1], thism, &y_colidx->data[0],
                &y_rowidx->data[0], &y_d->data[0]);
          }
          c_S = cs_di_sqr(2, c_cxA, 1);
          c_N = cs_di_qr(c_cxA, c_S);
          cs_di_spfree(c_cxA);
          qr_rank_di(c_N, &M);
          i = zik->size[0];
          zik->size[0] = cidxInt->data[cidxInt->size[0] - 1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(zik, i, &bd_emlrtRTEI);
          if (rhs->size[0] < cidxInt->data[cidxInt->size[0] - 1]) {
            i = b_outBuff->size[0];
            b_outBuff->size[0] = cidxInt->data[cidxInt->size[0] - 1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(b_outBuff, i, &dd_emlrtRTEI);
          } else {
            i = b_outBuff->size[0];
            b_outBuff->size[0] = rhs->size[0];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(b_outBuff, i, &cd_emlrtRTEI);
          }
          thism = rhs->size[0];
          profileLoopStart("getCoeffsAndInitialConditions_loop_37", __LINE__,
                           (thism - 1) + 1, "");
          for (i = 0; i < thism; i++) {
            b_outBuff->data[i] = rhs->data[i];
          }
          profileLoopEnd();
          solve_from_qr_di(c_N, c_S, (double *)&b_outBuff->data[0],
                           rhs->size[0], cidxInt->data[cidxInt->size[0] - 1]);
          cptr = cidxInt->data[cidxInt->size[0] - 1];
          profileLoopStart("getCoeffsAndInitialConditions_loop_38", __LINE__,
                           (cptr - 1) + 1, "");
          for (i = 0; i < cptr; i++) {
            zik->data[i] = b_outBuff->data[i];
          }
          profileLoopEnd();
          cs_di_sfree(c_S);
          cs_di_nfree(c_N);
        } else {
          i = zik->size[0];
          zik->size[0] = rhs->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(zik, i, &ad_emlrtRTEI);
          profileLoopStart("getCoeffsAndInitialConditions_loop_34", __LINE__,
                           (rhs->size[0] - 1) + 1, "");
          for (i = 0; i < rhs->size[0]; i++) {
            zik->data[i] = rhs->data[i];
          }
          profileLoopEnd();
          solve_from_lu_di(b_N, b_S, (double *)&zik->data[0], rhs->size[0]);
          cs_di_sfree(b_S);
          cs_di_nfree(b_N);
        }
      } else {
        cs_di *cxA;
        cs_din *N;
        cs_dis *S;
        if (thism < cidxInt->data[cidxInt->size[0] - 1]) {
          nvtxMarkA("#sparse_ctranspose#" MW_AT_LINE);
          sparse_ctranspose(y_d, y_colidx, y_rowidx, thism,
                            cidxInt->data[cidxInt->size[0] - 1], in_d,
                            in_colidx, in_rowidx, &cptr, &ny);
          cxA = makeCXSparseMatrix(in_colidx->data[in_colidx->size[0] - 1] - 1,
                                   ny, cptr, &in_colidx->data[0],
                                   &in_rowidx->data[0], &in_d->data[0]);
        } else {
          cxA = makeCXSparseMatrix(y_colidx->data[y_colidx->size[0] - 1] - 1,
                                   cidxInt->data[cidxInt->size[0] - 1], thism,
                                   &y_colidx->data[0], &y_rowidx->data[0],
                                   &y_d->data[0]);
        }
        S = cs_di_sqr(2, cxA, 1);
        N = cs_di_qr(cxA, S);
        cs_di_spfree(cxA);
        cptr = qr_rank_di(N, &M);
        if (thism > cidxInt->data[cidxInt->size[0] - 1]) {
          thism = cidxInt->data[cidxInt->size[0] - 1];
        }
        if (cptr < thism) {
          char_T str[14];
          y = nullptr;
          m = emlrtCreateCharArray(2, &iv[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 6, m, &rfmt[0]);
          emlrtAssign(&y, m);
          b_y = nullptr;
          m1 = emlrtCreateDoubleScalar(M);
          emlrtAssign(&b_y, m1);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(y, b_y, &d_emlrtMCI),
                           "<output of sprintf>", str);
          nvtxMarkA("#warning#" MW_AT_LINE);
          internal::warning(cptr, str);
        }
        i = zik->size[0];
        zik->size[0] = cidxInt->data[cidxInt->size[0] - 1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(zik, i, &bd_emlrtRTEI);
        if (rhs->size[0] < cidxInt->data[cidxInt->size[0] - 1]) {
          i = outBuff->size[0];
          outBuff->size[0] = cidxInt->data[cidxInt->size[0] - 1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(outBuff, i, &dd_emlrtRTEI);
        } else {
          i = outBuff->size[0];
          outBuff->size[0] = rhs->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(outBuff, i, &cd_emlrtRTEI);
        }
        thism = rhs->size[0];
        profileLoopStart("getCoeffsAndInitialConditions_loop_35", __LINE__,
                         (thism - 1) + 1, "");
        for (i = 0; i < thism; i++) {
          outBuff->data[i] = rhs->data[i];
        }
        profileLoopEnd();
        solve_from_qr_di(N, S, (double *)&outBuff->data[0], rhs->size[0],
                         cidxInt->data[cidxInt->size[0] - 1]);
        cptr = cidxInt->data[cidxInt->size[0] - 1];
        profileLoopStart("getCoeffsAndInitialConditions_loop_36", __LINE__,
                         (cptr - 1) + 1, "");
        for (i = 0; i < cptr; i++) {
          zik->data[i] = outBuff->data[i];
        }
        profileLoopEnd();
        cs_di_sfree(S);
        cs_di_nfree(N);
      }
      cptr = zi->size[0];
      profileLoopStart("getCoeffsAndInitialConditions_loop_33", __LINE__,
                       (cptr - 1) + 1, "");
      for (i = 0; i < cptr; i++) {
        zi->data[i + zi->size[0] * ns] = zik->data[i];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_outBuff);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&outBuff);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&c_in_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&c_in_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c_in_d);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_in_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_in_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&b_in_d);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&in_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&in_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&in_d);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&zik);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&b_t);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&t);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&sortedIndices);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&cidxInt);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&ridxInt);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&y_rowidx);
    nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
    emxFree_int32_T(&y_colidx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&y_d);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&vals);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&colIdx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&rowIdx);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&rhs);
  } else {
    zi->size[0] = 0;
    zi->size[1] = static_cast<int32_T>(numStage);
  }
  *Letr = std::fmax(1.0, 3.0 * ord);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

} // namespace coder
static int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  nvtxRangePushA("#fcn#div_s32#" MW_AT_LOCATION);
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  nvtxRangePop();
  return quotient;
}

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[14])
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[14])
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

//
//
namespace coder {
void filtfilt(const emxArray_real_T *ctf, const emxArray_real_T *varargin_1,
              const emxArray_real_T *varargin_2, emxArray_real_T *y)
{
  emxArray_real_T b_gpu_A;
  emxArray_real_T b_gpu_B;
  emxArray_real_T b_gpu_xt;
  emxArray_real_T b_gpu_yc2;
  emxArray_real_T b_gpu_yc3;
  emxArray_real_T b_gpu_yc5;
  emxArray_real_T b_gpu_ytemp;
  emxArray_real_T b_gpu_zi;
  emxArray_real_T b_gpu_zo;
  emxArray_real_T c_gpu_A;
  emxArray_real_T c_gpu_B;
  emxArray_real_T c_gpu_xt;
  emxArray_real_T c_gpu_yc2;
  emxArray_real_T c_gpu_yc3;
  emxArray_real_T c_gpu_ytemp;
  emxArray_real_T c_gpu_zi;
  emxArray_real_T c_gpu_zo;
  emxArray_real_T d_gpu_A;
  emxArray_real_T d_gpu_B;
  emxArray_real_T d_gpu_yc2;
  emxArray_real_T d_gpu_yc3;
  emxArray_real_T d_gpu_zi;
  emxArray_real_T d_gpu_zo;
  emxArray_real_T e_gpu_A;
  emxArray_real_T e_gpu_B;
  emxArray_real_T e_gpu_zi;
  emxArray_real_T e_gpu_zo;
  emxArray_real_T f_gpu_A;
  emxArray_real_T f_gpu_B;
  emxArray_real_T f_gpu_zi;
  emxArray_real_T f_gpu_zo;
  emxArray_real_T g_gpu_A;
  emxArray_real_T g_gpu_B;
  emxArray_real_T gpu_A;
  emxArray_real_T gpu_B;
  emxArray_real_T gpu__1;
  emxArray_real_T gpu__2;
  emxArray_real_T gpu__3;
  emxArray_real_T gpu__4;
  emxArray_real_T gpu_r2;
  emxArray_real_T gpu_xc1;
  emxArray_real_T gpu_xt;
  emxArray_real_T gpu_y;
  emxArray_real_T gpu_yc2;
  emxArray_real_T gpu_yc3;
  emxArray_real_T gpu_yc5;
  emxArray_real_T gpu_ytemp;
  emxArray_real_T gpu_zi;
  emxArray_real_T gpu_zo;
  emxArray_real_T h_gpu_A;
  emxArray_real_T h_gpu_B;
  emxArray_real_T i_gpu_A;
  emxArray_real_T i_gpu_B;
  emxArray_real_T j_gpu_A;
  emxArray_real_T j_gpu_B;
  emxArray_real_T k_gpu_A;
  emxArray_real_T k_gpu_B;
  emxArray_real_T l_gpu_A;
  emxArray_real_T l_gpu_B;
  emxArray_real_T *A;
  emxArray_real_T *B;
  emxArray_real_T *X;
  emxArray_real_T *Y;
  emxArray_real_T *b_cpu_A;
  emxArray_real_T *b_cpu_B;
  emxArray_real_T *b_cpu_xt;
  emxArray_real_T *b_cpu_yc2;
  emxArray_real_T *b_cpu_yc3;
  emxArray_real_T *b_cpu_yc5;
  emxArray_real_T *b_cpu_ytemp;
  emxArray_real_T *b_cpu_zi;
  emxArray_real_T *b_cpu_zo;
  emxArray_real_T *b_xc1;
  emxArray_real_T *c_cpu_A;
  emxArray_real_T *c_cpu_B;
  emxArray_real_T *c_cpu_xt;
  emxArray_real_T *c_cpu_yc2;
  emxArray_real_T *c_cpu_yc3;
  emxArray_real_T *c_cpu_ytemp;
  emxArray_real_T *c_cpu_zi;
  emxArray_real_T *c_cpu_zo;
  emxArray_real_T *cpu_A;
  emxArray_real_T *cpu_B;
  emxArray_real_T *cpu__1;
  emxArray_real_T *cpu__2;
  emxArray_real_T *cpu__3;
  emxArray_real_T *cpu__4;
  emxArray_real_T *cpu_r2;
  emxArray_real_T *cpu_xc1;
  emxArray_real_T *cpu_xt;
  emxArray_real_T *cpu_y;
  emxArray_real_T *cpu_yc2;
  emxArray_real_T *cpu_yc3;
  emxArray_real_T *cpu_yc5;
  emxArray_real_T *cpu_ytemp;
  emxArray_real_T *cpu_zi;
  emxArray_real_T *cpu_zo;
  emxArray_real_T *d_cpu_A;
  emxArray_real_T *d_cpu_B;
  emxArray_real_T *d_cpu_yc2;
  emxArray_real_T *d_cpu_yc3;
  emxArray_real_T *d_cpu_zi;
  emxArray_real_T *d_cpu_zo;
  emxArray_real_T *den;
  emxArray_real_T *e_cpu_A;
  emxArray_real_T *e_cpu_B;
  emxArray_real_T *e_cpu_zi;
  emxArray_real_T *e_cpu_zo;
  emxArray_real_T *f_cpu_A;
  emxArray_real_T *f_cpu_B;
  emxArray_real_T *f_cpu_zi;
  emxArray_real_T *f_cpu_zo;
  emxArray_real_T *g_cpu_A;
  emxArray_real_T *g_cpu_B;
  emxArray_real_T *h_cpu_A;
  emxArray_real_T *h_cpu_B;
  emxArray_real_T *i_cpu_A;
  emxArray_real_T *i_cpu_B;
  emxArray_real_T *j_cpu_A;
  emxArray_real_T *j_cpu_B;
  emxArray_real_T *k_cpu_A;
  emxArray_real_T *k_cpu_B;
  emxArray_real_T *l_cpu_A;
  emxArray_real_T *l_cpu_B;
  emxArray_real_T *num;
  emxArray_real_T *r;
  emxArray_real_T *r1;
  emxArray_real_T *xc1;
  emxArray_real_T *xt;
  emxArray_real_T *ytemp;
  emxArray_real_T *zi;
  real_T Letr;
  real_T P;
  real_T Q;
  real_T numStage;
  int32_T i;
  boolean_T A_outdatedOnCpu;
  boolean_T A_outdatedOnGpu;
  boolean_T B_outdatedOnCpu;
  boolean_T B_outdatedOnGpu;
  boolean_T a__2_outdatedOnGpu;
  boolean_T a__4_outdatedOnGpu;
  boolean_T b_zo_outdatedOnCpu;
  boolean_T b_zo_outdatedOnGpu;
  boolean_T r2_outdatedOnCpu;
  boolean_T xIsRow;
  boolean_T xt_outdatedOnCpu;
  boolean_T xt_outdatedOnGpu;
  boolean_T yc2_outdatedOnCpu;
  boolean_T yc3_outdatedOnGpu;
  boolean_T yc5_outdatedOnCpu;
  boolean_T yc5_outdatedOnGpu;
  boolean_T zi_outdatedOnCpu;
  boolean_T zi_outdatedOnGpu;
  boolean_T zo_outdatedOnCpu;
  boolean_T zo_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#filtfilt#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_ytemp);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_r2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&g_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&g_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&l_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&l_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&f_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&f_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&f_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&k_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&k_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_xt);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&e_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&e_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&j_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&j_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_zi);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xt);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&i_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&i_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&e_gpu_zi);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&h_gpu_A);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&h_gpu_B);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_yc5);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_yc5);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu__2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&f_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu__4);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_yc3);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_ytemp);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&e_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_yc2);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_zo);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu__3);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_zo);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu__1);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_xt);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xc1);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_ytemp);
  r2_outdatedOnCpu = false;
  yc3_outdatedOnGpu = false;
  b_zo_outdatedOnCpu = false;
  b_zo_outdatedOnGpu = false;
  yc2_outdatedOnCpu = false;
  yc5_outdatedOnCpu = false;
  zo_outdatedOnCpu = false;
  zo_outdatedOnGpu = false;
  yc5_outdatedOnGpu = false;
  a__2_outdatedOnGpu = false;
  xt_outdatedOnCpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&X, 3, &b_emlrtRTEI, true);
  i = X->size[0] * X->size[1] * X->size[2];
  X->size[0] = varargin_2->size[0];
  X->size[1] = varargin_2->size[1];
  X->size[2] = varargin_2->size[2];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(X, i, &emlrtRTEI);
  profileLoopStart(
      "filtfilt_loop_0", __LINE__,
      (varargin_2->size[0] * varargin_2->size[1] * varargin_2->size[2] - 1) + 1,
      "1675");
  for (i = 0;
       i < varargin_2->size[0] * varargin_2->size[1] * varargin_2->size[2];
       i++) {
    X->data[i] = varargin_2->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&num, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&den, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#filtfiltParser#" MW_AT_LINE);
  filtfiltParser(ctf, varargin_1, num, den, &numStage, &P, &Q, &xIsRow,
                 &yc5_outdatedOnGpu);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&B, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&A, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&zi, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&Y, 1, &i_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_ytemp, 1, &ac_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xc1, 2, &bc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&xt, 1, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xt, 2, &cc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu__1, 1, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_zo, 1, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu__3, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_zo, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_yc2, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_zo, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_yc2, 1, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_zo, 1, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_ytemp, 1, &dc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_yc3, 2, &ec_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_yc3, 1, &fc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu__4, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&e_cpu_zo, 2, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu__2, 1, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&f_cpu_zo, 1, &yb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_yc5, 2, &gc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_yc5, 1, &hc_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&r, 2, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&xc1, 2, &j_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_B, 2, &m_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_A, 2, &o_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_zi, 2, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_B, 2, &u_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_A, 2, &v_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_B, 2, &n_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_A, 2, &p_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_xt, 1, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_zi, 1, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_B, 2, &w_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_A, 2, &y_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_y, 1, &bb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&e_cpu_B, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&e_cpu_A, 2, &t_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_zi, 1, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&ytemp, 1, &x_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&f_cpu_B, 2, &cb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&f_cpu_A, 2, &eb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_zi, 1, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&r1, 2, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_xc1, 2, &db_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&g_cpu_B, 2, &fb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&g_cpu_A, 2, &gb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&h_cpu_B, 2, &kb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&h_cpu_A, 2, &mb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_xt, 1, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&i_cpu_B, 2, &jb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&i_cpu_A, 2, &lb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_yc3, 2, &nb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&e_cpu_zi, 2, &ob_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&j_cpu_B, 2, &pb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&j_cpu_A, 2, &rb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_yc3, 1, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&f_cpu_zi, 1, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&k_cpu_B, 2, &qb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&k_cpu_A, 2, &sb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_yc2, 2, &tb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&l_cpu_B, 2, &ub_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&l_cpu_A, 2, &wb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_yc2, 1, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_r2, 2, &ic_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_ytemp, 1, &hb_emlrtRTEI, true);
  if ((!xIsRow) || (!yc5_outdatedOnGpu) ||
      ((X->size[0] == 0) || (X->size[1] == 0) || (X->size[2] == 0))) {
    y->size[0] = 0;
    y->size[1] = 0;
    y->size[2] = 1;
  } else {
    if ((X->size[2] == 1) && (X->size[0] == 1)) {
      i = X->size[0] * X->size[1] * X->size[2];
      X->size[0] = X->size[1];
      X->size[1] = 1;
      X->size[2] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(X, i, &b_emlrtRTEI);
    } else {
      xIsRow = false;
    }
    nvtxMarkA("#getCoeffsAndInitialConditions#" MW_AT_LINE);
    getCoeffsAndInitialConditions(num, den, numStage, P, Q, B, A, zi, &Letr);
    if (X->size[1] * X->size[2] == 1) {
      if (X->size[0] < 10000) {
        int32_T i1;
        yc5_outdatedOnGpu = false;
        a__2_outdatedOnGpu = false;
        i = Y->size[0];
        Y->size[0] = X->size[0];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(Y, i, &e_emlrtRTEI);
        profileLoopStart("filtfilt_loop_2", __LINE__, (X->size[0] - 1) + 1,
                         "1676");
        for (i = 0; i < X->size[0]; i++) {
          Y->data[i] = X->data[i];
        }
        profileLoopEnd();
        i1 = static_cast<int32_T>(numStage);
        profileLoopStart("filtfilt_loop_5", __LINE__, (i1 - 1) + 1, "1677");
        for (int32_T ii{0}; ii < i1; ii++) {
          int32_T b_loop_ub;
          int32_T i10;
          int32_T i8;
          int32_T loop_ub;
          numStage = static_cast<real_T>(Y->size[0]) - Letr;
          if (numStage > static_cast<real_T>(Y->size[0]) - 1.0) {
            loop_ub = 0;
            i8 = 1;
            i10 = -1;
          } else {
            loop_ub = Y->size[0] - 2;
            i8 = -1;
            i10 = static_cast<int32_T>(numStage) - 1;
          }
          numStage = 2.0 * Y->data[0];
          P = 2.0 * Y->data[Y->size[0] - 1];
          yc5_outdatedOnCpu = false;
          i = b_cpu_ytemp->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          b_cpu_ytemp->size[0] =
              (static_cast<int32_T>(Letr + 1.0) + Y->size[0]) +
              div_s32(i10 - loop_ub, i8);
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_cpu_ytemp, i, &l_emlrtRTEI);
          b_loop_ub = static_cast<int32_T>(Letr + 1.0) - 2;
          profileLoopStart("filtfilt_loop_11", __LINE__, b_loop_ub + 1, "1678");
          for (i = 0; i <= b_loop_ub; i++) {
            b_cpu_ytemp->data[i] =
                numStage - Y->data[(static_cast<int32_T>(Letr + 1.0) - i) - 1];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          b_loop_ub = Y->size[0];
          profileLoopStart("filtfilt_loop_14", __LINE__, (b_loop_ub - 1) + 1,
                           "1679");
          for (i = 0; i < b_loop_ub; i++) {
            b_cpu_ytemp->data[(i + static_cast<int32_T>(Letr + 1.0)) - 1] =
                Y->data[i];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          b_loop_ub = div_s32(i10 - loop_ub, i8);
          profileLoopStart("filtfilt_loop_15", __LINE__, b_loop_ub + 1, "1680");
          for (i = 0; i <= b_loop_ub; i++) {
            b_cpu_ytemp
                ->data[((i + static_cast<int32_T>(Letr + 1.0)) + Y->size[0]) -
                       1] = P - Y->data[loop_ub + i8 * i];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = e_cpu_B->size[0] * e_cpu_B->size[1];
          e_cpu_B->size[0] = 1;
          e_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(e_cpu_B, i, &q_emlrtRTEI);
          profileLoopStart("filtfilt_loop_19", __LINE__, (B->size[1] - 1) + 1,
                           "1681");
          for (i = 0; i < B->size[1]; i++) {
            e_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = e_cpu_A->size[0] * e_cpu_A->size[1];
          e_cpu_A->size[0] = 1;
          e_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(e_cpu_A, i, &t_emlrtRTEI);
          profileLoopStart("filtfilt_loop_22", __LINE__, (A->size[1] - 1) + 1,
                           "1682");
          for (i = 0; i < A->size[1]; i++) {
            e_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = c_cpu_zi->size[0];
          c_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(c_cpu_zi, i, &f_emlrtRTEI);
          profileLoopStart("filtfilt_loop_24", __LINE__, (zi->size[0] - 1) + 1,
                           "1683");
          for (i = 0; i < zi->size[0]; i++) {
            c_cpu_zi->data[i] =
                zi->data[i + zi->size[0] * ii] * b_cpu_ytemp->data[0];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(e_cpu_B, &B_outdatedOnCpu, &gpu_B, &B_outdatedOnGpu, e_cpu_A,
                 &A_outdatedOnCpu, &gpu_A, &A_outdatedOnGpu, b_cpu_ytemp,
                 &gpu_ytemp, &yc5_outdatedOnCpu, c_cpu_zi, &zi_outdatedOnCpu,
                 &gpu_zi, &zi_outdatedOnGpu, cpu_ytemp, &yc5_outdatedOnGpu,
                 &b_gpu_ytemp, &a__2_outdatedOnGpu);
          loop_ub = cpu_ytemp->size[0] - 1;
          i = ytemp->size[0];
          ytemp->size[0] = cpu_ytemp->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(ytemp, i, &x_emlrtRTEI);
          profileLoopStart("filtfilt_loop_28", __LINE__, loop_ub + 1, "1684");
          for (i = 0; i <= loop_ub; i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(cpu_ytemp, &b_gpu_ytemp);
            }
            yc5_outdatedOnGpu = false;
            ytemp->data[i] = cpu_ytemp->data[loop_ub - i];
          }
          profileLoopEnd();
          yc5_outdatedOnGpu = false;
          a__2_outdatedOnGpu = false;
          i = cpu_ytemp->size[0];
          cpu_ytemp->size[0] = ytemp->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_ytemp, i, &ab_emlrtRTEI);
          profileLoopStart("filtfilt_loop_30", __LINE__,
                           (ytemp->size[0] - 1) + 1, "1685");
          for (i = 0; i < ytemp->size[0]; i++) {
            cpu_ytemp->data[i] = ytemp->data[i];
            a__2_outdatedOnGpu = true;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = f_cpu_B->size[0] * f_cpu_B->size[1];
          f_cpu_B->size[0] = 1;
          f_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(f_cpu_B, i, &cb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_33", __LINE__, (B->size[1] - 1) + 1,
                           "1686");
          for (i = 0; i < B->size[1]; i++) {
            f_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = f_cpu_A->size[0] * f_cpu_A->size[1];
          f_cpu_A->size[0] = 1;
          f_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(f_cpu_A, i, &eb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_35", __LINE__, (A->size[1] - 1) + 1,
                           "1687");
          for (i = 0; i < A->size[1]; i++) {
            f_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = d_cpu_zi->size[0];
          d_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(d_cpu_zi, i, &f_emlrtRTEI);
          profileLoopStart("filtfilt_loop_39", __LINE__, (zi->size[0] - 1) + 1,
                           "1688");
          for (i = 0; i < zi->size[0]; i++) {
            d_cpu_zi->data[i] =
                zi->data[i + zi->size[0] * ii] * cpu_ytemp->data[0];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          yc5_outdatedOnCpu = false;
          i = c_cpu_ytemp->size[0];
          c_cpu_ytemp->size[0] = cpu_ytemp->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(c_cpu_ytemp, i, &hb_emlrtRTEI);
          b_loop_ub = cpu_ytemp->size[0] - 1;
          profileLoopStart("filtfilt_loop_42", __LINE__, b_loop_ub + 1, "1689");
          for (i = 0; i <= b_loop_ub; i++) {
            c_cpu_ytemp->data[i] = cpu_ytemp->data[i];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(f_cpu_B, &B_outdatedOnCpu, &b_gpu_B, &B_outdatedOnGpu, f_cpu_A,
                 &A_outdatedOnCpu, &b_gpu_A, &A_outdatedOnGpu, c_cpu_ytemp,
                 &c_gpu_ytemp, &yc5_outdatedOnCpu, d_cpu_zi, &zi_outdatedOnCpu,
                 &b_gpu_zi, &zi_outdatedOnGpu, cpu_ytemp, &yc5_outdatedOnGpu,
                 &b_gpu_ytemp, &a__2_outdatedOnGpu);
          numStage = static_cast<real_T>(cpu_ytemp->size[0]) - Letr;
          if (Letr + 1.0 > numStage) {
            loop_ub = 1;
            i8 = 1;
          } else {
            loop_ub = static_cast<int32_T>(numStage);
            i8 = -1;
          }
          i10 = Y->size[0];
          i = Y->size[0];
          Y->size[0] = i10;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(Y, i, &e_emlrtRTEI);
          profileLoopStart("filtfilt_loop_45", __LINE__, (i10 - 1) + 1, "1690");
          for (i = 0; i < i10; i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(cpu_ytemp, &b_gpu_ytemp);
            }
            yc5_outdatedOnGpu = false;
            Y->data[i] = cpu_ytemp->data[(loop_ub + i8 * i) - 1];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        i = y->size[0] * y->size[1] * y->size[2];
        y->size[0] = Y->size[0];
        y->size[1] = 1;
        y->size[2] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, i, &i_emlrtRTEI);
        profileLoopStart("filtfilt_loop_8", __LINE__, (Y->size[0] - 1) + 1,
                         "1691");
        for (i = 0; i < Y->size[0]; i++) {
          y->data[i] = Y->data[i];
        }
        profileLoopEnd();
      } else {
        int32_T i1;
        i = y->size[0] * y->size[1] * y->size[2];
        y->size[0] = X->size[0];
        y->size[1] = X->size[1];
        y->size[2] = X->size[2];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, i, &c_emlrtRTEI);
        profileLoopStart("filtfilt_loop_1", __LINE__,
                         (X->size[0] * X->size[1] * X->size[2] - 1) + 1,
                         "1692");
        for (i = 0; i < X->size[0] * X->size[1] * X->size[2]; i++) {
          y->data[i] = X->data[i];
        }
        profileLoopEnd();
        i1 = static_cast<int32_T>(numStage);
        profileLoopStart("filtfilt_loop_4", __LINE__, (i1 - 1) + 1, "1693");
        for (int32_T ii{0}; ii < i1; ii++) {
          int32_T b_i1;
          int32_T b_loop_ub;
          int32_T i10;
          int32_T i8;
          int32_T loop_ub;
          numStage = 2.0 * y->data[0];
          i = static_cast<int32_T>(Letr + 1.0) - 1;
          b_i1 = xt->size[0];
          xt->size[0] = static_cast<int32_T>(Letr + 1.0) - 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(xt, b_i1, &k_emlrtRTEI);
          profileLoopStart("filtfilt_loop_7", __LINE__, (i - 1) + 1, "1694");
          for (b_i1 = 0; b_i1 < i; b_i1++) {
            xt->data[b_i1] =
                -y->data[(static_cast<int32_T>(Letr + 1.0) - b_i1) - 1] +
                numStage;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = c_cpu_B->size[0] * c_cpu_B->size[1];
          c_cpu_B->size[0] = 1;
          c_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(c_cpu_B, i, &n_emlrtRTEI);
          profileLoopStart("filtfilt_loop_12", __LINE__, (B->size[1] - 1) + 1,
                           "1695");
          for (i = 0; i < B->size[1]; i++) {
            c_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = c_cpu_A->size[0] * c_cpu_A->size[1];
          c_cpu_A->size[0] = 1;
          c_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(c_cpu_A, i, &p_emlrtRTEI);
          profileLoopStart("filtfilt_loop_16", __LINE__, (A->size[1] - 1) + 1,
                           "1696");
          for (i = 0; i < A->size[1]; i++) {
            c_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          xt_outdatedOnGpu = false;
          i = b_cpu_xt->size[0];
          b_cpu_xt->size[0] = xt->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_cpu_xt, i, &s_emlrtRTEI);
          profileLoopStart("filtfilt_loop_18", __LINE__, (xt->size[0] - 1) + 1,
                           "1697");
          for (i = 0; i < xt->size[0]; i++) {
            b_cpu_xt->data[i] = xt->data[i];
            xt_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = b_cpu_zi->size[0];
          b_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_cpu_zi, i, &f_emlrtRTEI);
          profileLoopStart("filtfilt_loop_21", __LINE__, (zi->size[0] - 1) + 1,
                           "1698");
          for (i = 0; i < zi->size[0]; i++) {
            b_cpu_zi->data[i] = zi->data[i + zi->size[0] * ii] * xt->data[0];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#b_filter#" MW_AT_LINE);
          b_filter(c_cpu_B, &B_outdatedOnCpu, &c_gpu_B, &B_outdatedOnGpu,
                   c_cpu_A, &A_outdatedOnCpu, &c_gpu_A, &A_outdatedOnGpu,
                   b_cpu_xt, &gpu_xt, &xt_outdatedOnGpu, b_cpu_zi,
                   &zi_outdatedOnCpu, &c_gpu_zi, &zi_outdatedOnGpu, cpu__1,
                   &yc5_outdatedOnGpu, &gpu__1, &a__2_outdatedOnGpu, cpu_zo,
                   &zo_outdatedOnCpu, &gpu_zo, &zo_outdatedOnGpu);
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = d_cpu_B->size[0] * d_cpu_B->size[1];
          d_cpu_B->size[0] = 1;
          d_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(d_cpu_B, i, &w_emlrtRTEI);
          profileLoopStart("filtfilt_loop_26", __LINE__, (B->size[1] - 1) + 1,
                           "1699");
          for (i = 0; i < B->size[1]; i++) {
            d_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = d_cpu_A->size[0] * d_cpu_A->size[1];
          d_cpu_A->size[0] = 1;
          d_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(d_cpu_A, i, &y_emlrtRTEI);
          profileLoopStart("filtfilt_loop_29", __LINE__, (A->size[1] - 1) + 1,
                           "1700");
          for (i = 0; i < A->size[1]; i++) {
            d_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          i = y->size[0];
          yc5_outdatedOnCpu = false;
          b_i1 = cpu_y->size[0];
          cpu_y->size[0] = y->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_y, b_i1, &bb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_32", __LINE__, (i - 1) + 1, "1701");
          for (b_i1 = 0; b_i1 < i; b_i1++) {
            cpu_y->data[b_i1] = y->data[b_i1];
            yc5_outdatedOnCpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#b_filter#" MW_AT_LINE);
          b_filter(d_cpu_B, &B_outdatedOnCpu, &d_gpu_B, &B_outdatedOnGpu,
                   d_cpu_A, &A_outdatedOnCpu, &d_gpu_A, &A_outdatedOnGpu, cpu_y,
                   &gpu_y, &yc5_outdatedOnCpu, cpu_zo, &zo_outdatedOnCpu,
                   &gpu_zo, &zo_outdatedOnGpu, b_cpu_yc2, &yc2_outdatedOnCpu,
                   &gpu_yc2, &a__2_outdatedOnGpu, d_cpu_zo, &b_zo_outdatedOnCpu,
                   &b_gpu_zo, &b_zo_outdatedOnGpu);
          numStage = static_cast<real_T>(y->size[0]) - Letr;
          if (numStage > static_cast<real_T>(y->size[0]) - 1.0) {
            loop_ub = 0;
            i8 = 1;
            i10 = -1;
          } else {
            loop_ub = y->size[0] - 2;
            i8 = -1;
            i10 = static_cast<int32_T>(numStage) - 1;
          }
          numStage = 2.0 * y->data[y->size[0] - 1];
          i = xt->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          xt->size[0] = div_s32(i10 - loop_ub, i8) + 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(xt, i, &ib_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_34", __LINE__,
                           div_s32(i10 - loop_ub, i8) + 1, "1702");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (i = 0; i <= div_s32(i10 - loop_ub, i8); i++) {
            xt->data[i] = -y->data[loop_ub + i8 * i] + numStage;
          }
          profileLoopEnd();
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = h_cpu_B->size[0] * h_cpu_B->size[1];
          h_cpu_B->size[0] = 1;
          h_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(h_cpu_B, i, &kb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_38", __LINE__, (B->size[1] - 1) + 1,
                           "1703");
          for (i = 0; i < B->size[1]; i++) {
            h_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = h_cpu_A->size[0] * h_cpu_A->size[1];
          h_cpu_A->size[0] = 1;
          h_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(h_cpu_A, i, &mb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_41", __LINE__, (A->size[1] - 1) + 1,
                           "1704");
          for (i = 0; i < A->size[1]; i++) {
            h_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          xt_outdatedOnGpu = false;
          i = c_cpu_xt->size[0];
          c_cpu_xt->size[0] = xt->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(c_cpu_xt, i, &s_emlrtRTEI);
          profileLoopStart("filtfilt_loop_44", __LINE__, (xt->size[0] - 1) + 1,
                           "1705");
          for (i = 0; i < xt->size[0]; i++) {
            c_cpu_xt->data[i] = xt->data[i];
            xt_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(h_cpu_B, &B_outdatedOnCpu, &e_gpu_B, &B_outdatedOnGpu, h_cpu_A,
                 &A_outdatedOnCpu, &e_gpu_A, &A_outdatedOnGpu, c_cpu_xt,
                 &b_gpu_xt, &xt_outdatedOnGpu, d_cpu_zo, &b_zo_outdatedOnCpu,
                 &b_gpu_zo, &b_zo_outdatedOnGpu, b_cpu_yc3, &yc5_outdatedOnGpu,
                 &gpu_yc3, &yc3_outdatedOnGpu);
          if (b_cpu_yc3->size[0] < 1) {
            loop_ub = 0;
            i8 = 1;
            i10 = -1;
          } else {
            loop_ub = b_cpu_yc3->size[0] - 1;
            i8 = -1;
            i10 = 0;
          }
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = j_cpu_B->size[0] * j_cpu_B->size[1];
          j_cpu_B->size[0] = 1;
          j_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(j_cpu_B, i, &pb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_46", __LINE__, (B->size[1] - 1) + 1,
                           "1706");
          for (i = 0; i < B->size[1]; i++) {
            j_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = j_cpu_A->size[0] * j_cpu_A->size[1];
          j_cpu_A->size[0] = 1;
          j_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(j_cpu_A, i, &rb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_48", __LINE__, (A->size[1] - 1) + 1,
                           "1707");
          for (i = 0; i < A->size[1]; i++) {
            j_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          yc5_outdatedOnCpu = false;
          yc3_outdatedOnGpu = false;
          i = d_cpu_yc3->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          d_cpu_yc3->size[0] = div_s32(i10 - loop_ub, i8) + 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(d_cpu_yc3, i, &s_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_50", __LINE__,
                           div_s32(i10 - loop_ub, i8) + 1, "1708");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (i = 0; i <= div_s32(i10 - loop_ub, i8); i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(b_cpu_yc3, &gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            d_cpu_yc3->data[i] = b_cpu_yc3->data[loop_ub + i8 * i];
            yc3_outdatedOnGpu = true;
          }
          profileLoopEnd();
          zi_outdatedOnCpu = false;
          zi_outdatedOnGpu = false;
          i = f_cpu_zi->size[0];
          f_cpu_zi->size[0] = zi->size[0];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(f_cpu_zi, i, &f_emlrtRTEI);
          profileLoopStart("filtfilt_loop_52", __LINE__, (zi->size[0] - 1) + 1,
                           "1709");
          for (i = 0; i < zi->size[0]; i++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(b_cpu_yc3, &gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            f_cpu_zi->data[i] = zi->data[i + zi->size[0] * ii] *
                                b_cpu_yc3->data[b_cpu_yc3->size[0] - 1];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#b_filter#" MW_AT_LINE);
          b_filter(j_cpu_B, &B_outdatedOnCpu, &f_gpu_B, &B_outdatedOnGpu,
                   j_cpu_A, &A_outdatedOnCpu, &f_gpu_A, &A_outdatedOnGpu,
                   d_cpu_yc3, &b_gpu_yc3, &yc3_outdatedOnGpu, f_cpu_zi,
                   &zi_outdatedOnCpu, &d_gpu_zi, &zi_outdatedOnGpu, cpu__2,
                   &yc5_outdatedOnGpu, &gpu__2, &a__2_outdatedOnGpu, f_cpu_zo,
                   &b_zo_outdatedOnCpu, &c_gpu_zo, &b_zo_outdatedOnGpu);
          if (b_cpu_yc2->size[0] < 1) {
            loop_ub = 0;
            i8 = 1;
            i10 = -1;
          } else {
            loop_ub = b_cpu_yc2->size[0] - 1;
            i8 = -1;
            i10 = 0;
          }
          B_outdatedOnCpu = false;
          B_outdatedOnGpu = false;
          i = l_cpu_B->size[0] * l_cpu_B->size[1];
          l_cpu_B->size[0] = 1;
          l_cpu_B->size[1] = B->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(l_cpu_B, i, &ub_emlrtRTEI);
          profileLoopStart("filtfilt_loop_56", __LINE__, (B->size[1] - 1) + 1,
                           "1710");
          for (i = 0; i < B->size[1]; i++) {
            l_cpu_B->data[i] = B->data[ii + B->size[0] * i];
            B_outdatedOnGpu = true;
          }
          profileLoopEnd();
          A_outdatedOnCpu = false;
          A_outdatedOnGpu = false;
          i = l_cpu_A->size[0] * l_cpu_A->size[1];
          l_cpu_A->size[0] = 1;
          l_cpu_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(l_cpu_A, i, &wb_emlrtRTEI);
          profileLoopStart("filtfilt_loop_58", __LINE__, (A->size[1] - 1) + 1,
                           "1711");
          for (i = 0; i < A->size[1]; i++) {
            l_cpu_A->data[i] = A->data[ii + A->size[0] * i];
            A_outdatedOnGpu = true;
          }
          profileLoopEnd();
          a__2_outdatedOnGpu = false;
          i = d_cpu_yc2->size[0];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          d_cpu_yc2->size[0] = div_s32(i10 - loop_ub, i8) + 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(d_cpu_yc2, i, &s_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_59", __LINE__,
                           div_s32(i10 - loop_ub, i8) + 1, "1712");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (i = 0; i <= div_s32(i10 - loop_ub, i8); i++) {
            if (yc2_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(b_cpu_yc2, &gpu_yc2);
            }
            yc2_outdatedOnCpu = false;
            d_cpu_yc2->data[i] = b_cpu_yc2->data[loop_ub + i8 * i];
            a__2_outdatedOnGpu = true;
          }
          profileLoopEnd();
          nvtxMarkA("#filter#" MW_AT_LINE);
          filter(l_cpu_B, &B_outdatedOnCpu, &g_gpu_B, &B_outdatedOnGpu, l_cpu_A,
                 &A_outdatedOnCpu, &g_gpu_A, &A_outdatedOnGpu, d_cpu_yc2,
                 &b_gpu_yc2, &a__2_outdatedOnGpu, f_cpu_zo, &b_zo_outdatedOnCpu,
                 &c_gpu_zo, &b_zo_outdatedOnGpu, b_cpu_yc5, &yc5_outdatedOnCpu,
                 &gpu_yc5, &yc5_outdatedOnGpu);
          if (b_cpu_yc5->size[0] < 1) {
            i8 = 0;
            i10 = 1;
            loop_ub = -1;
          } else {
            i8 = b_cpu_yc5->size[0] - 1;
            i10 = -1;
            loop_ub = 0;
          }
          i = y->size[0] * y->size[1] * y->size[2];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          y->size[0] = div_s32(loop_ub - i8, i10) + 1;
          y->size[1] = 1;
          y->size[2] = 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(y, i, &xb_emlrtRTEI);
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          b_loop_ub = div_s32(loop_ub - i8, i10) + 1;
          profileLoopStart("filtfilt_loop_62", __LINE__, (b_loop_ub - 1) + 1,
                           "1713");
          for (i = 0; i < b_loop_ub; i++) {
            if (yc5_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(b_cpu_yc5, &gpu_yc5);
            }
            yc5_outdatedOnCpu = false;
            y->data[i] = b_cpu_yc5->data[i8 + i10 * i];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
      }
    } else {
      int32_T i1;
      uint32_T sz_idx_0;
      uint32_T sz_idx_1;
      uint32_T sz_idx_2;
      sz_idx_0 = static_cast<uint32_T>(X->size[0]);
      sz_idx_1 = static_cast<uint32_T>(X->size[1]);
      sz_idx_2 = static_cast<uint32_T>(X->size[2]);
      a__4_outdatedOnGpu = false;
      yc3_outdatedOnGpu = false;
      i = cpu_xc1->size[0] * cpu_xc1->size[1];
      cpu_xc1->size[0] = X->size[0];
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      cpu_xc1->size[1] =
          div_s32(X->size[0] * X->size[1] * X->size[2], X->size[0]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_xc1, i, &d_emlrtRTEI);
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      profileLoopStart("filtfilt_loop_3", __LINE__,
                       (static_cast<int32_T>(sz_idx_0) *
                            div_s32(X->size[0] * X->size[1] * X->size[2],
                                    static_cast<int32_T>(sz_idx_0)) -
                        1) +
                           1,
                       "1714");
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      for (i = 0; i < static_cast<int32_T>(sz_idx_0) *
                          div_s32(X->size[0] * X->size[1] * X->size[2],
                                  static_cast<int32_T>(sz_idx_0));
           i++) {
        cpu_xc1->data[i] = X->data[i];
        yc3_outdatedOnGpu = true;
      }
      profileLoopEnd();
      i1 = static_cast<int32_T>(numStage);
      profileLoopStart("filtfilt_loop_6", __LINE__, (i1 - 1) + 1, "1715");
      for (int32_T ii{0}; ii < i1; ii++) {
        int32_T b_i1;
        int32_T b_loop_ub;
        int32_T i10;
        int32_T i8;
        int32_T loop_ub;
        i = r->size[0] * r->size[1];
        r->size[0] = 1;
        r->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(r, i, &f_emlrtRTEI);
        i = static_cast<int32_T>(Letr + 1.0) - 1;
        b_i1 = xc1->size[0] * xc1->size[1];
        xc1->size[0] = static_cast<int32_T>(Letr + 1.0) - 1;
        xc1->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(xc1, b_i1, &j_emlrtRTEI);
        profileLoopStart("filtfilt_loop_10", __LINE__,
                         (cpu_xc1->size[1] - 1) + 1, "1716");
        for (b_i1 = 0; b_i1 < cpu_xc1->size[1]; b_i1++) {
          r->data[b_i1] = 2.0 * cpu_xc1->data[cpu_xc1->size[0] * b_i1];
          profileLoopStart("filtfilt_loop_13", __LINE__, (i - 1) + 1, "1717");
          for (loop_ub = 0; loop_ub < i; loop_ub++) {
            xc1->data[loop_ub + xc1->size[0] * b_i1] =
                cpu_xc1->data[((static_cast<int32_T>(Letr + 1.0) - loop_ub) +
                               cpu_xc1->size[0] * b_i1) -
                              1];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        if (xt_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_xt, &c_gpu_xt);
        }
        nvtxMarkA("#bsxfun#" MW_AT_LINE);
        bsxfun(r, xc1, cpu_xt);
        xt_outdatedOnCpu = false;
        xt_outdatedOnGpu = true;
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = cpu_B->size[0] * cpu_B->size[1];
        cpu_B->size[0] = 1;
        cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_B, i, &m_emlrtRTEI);
        profileLoopStart("filtfilt_loop_17", __LINE__, (B->size[1] - 1) + 1,
                         "1718");
        for (i = 0; i < B->size[1]; i++) {
          cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = cpu_A->size[0] * cpu_A->size[1];
        cpu_A->size[0] = 1;
        cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_A, i, &o_emlrtRTEI);
        profileLoopStart("filtfilt_loop_20", __LINE__, (A->size[1] - 1) + 1,
                         "1719");
        for (i = 0; i < A->size[1]; i++) {
          cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        zi_outdatedOnCpu = false;
        zi_outdatedOnGpu = false;
        i = cpu_zi->size[0] * cpu_zi->size[1];
        cpu_zi->size[0] = zi->size[0];
        cpu_zi->size[1] = cpu_xt->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_zi, i, &r_emlrtRTEI);
        b_loop_ub = cpu_xt->size[1];
        profileLoopStart("filtfilt_loop_23", __LINE__, (b_loop_ub - 1) + 1,
                         "1720");
        for (i = 0; i < b_loop_ub; i++) {
          loop_ub = zi->size[0];
          profileLoopStart("filtfilt_loop_25", __LINE__, (loop_ub - 1) + 1,
                           "1721");
          for (b_i1 = 0; b_i1 < loop_ub; b_i1++) {
            cpu_zi->data[b_i1 + cpu_zi->size[0] * i] =
                zi->data[b_i1 + zi->size[0] * ii] *
                cpu_xt->data[cpu_xt->size[0] * i];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        nvtxMarkA("#c_filter#" MW_AT_LINE);
        c_filter(cpu_B, &B_outdatedOnCpu, &h_gpu_B, &B_outdatedOnGpu, cpu_A,
                 &A_outdatedOnCpu, &h_gpu_A, &A_outdatedOnGpu, cpu_xt,
                 &c_gpu_xt, &xt_outdatedOnGpu, cpu_zi, &zi_outdatedOnCpu,
                 &e_gpu_zi, &zi_outdatedOnGpu, cpu__3, &yc5_outdatedOnGpu,
                 &gpu__3, &yc5_outdatedOnCpu, b_cpu_zo, &zo_outdatedOnCpu,
                 &d_gpu_zo, &zo_outdatedOnGpu);
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = b_cpu_B->size[0] * b_cpu_B->size[1];
        b_cpu_B->size[0] = 1;
        b_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_B, i, &u_emlrtRTEI);
        profileLoopStart("filtfilt_loop_27", __LINE__, (B->size[1] - 1) + 1,
                         "1722");
        for (i = 0; i < B->size[1]; i++) {
          b_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = b_cpu_A->size[0] * b_cpu_A->size[1];
        b_cpu_A->size[0] = 1;
        b_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_A, i, &v_emlrtRTEI);
        profileLoopStart("filtfilt_loop_31", __LINE__, (A->size[1] - 1) + 1,
                         "1723");
        for (i = 0; i < A->size[1]; i++) {
          b_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        nvtxMarkA("#c_filter#" MW_AT_LINE);
        c_filter(b_cpu_B, &B_outdatedOnCpu, &i_gpu_B, &B_outdatedOnGpu, b_cpu_A,
                 &A_outdatedOnCpu, &i_gpu_A, &A_outdatedOnGpu, cpu_xc1,
                 &gpu_xc1, &yc3_outdatedOnGpu, b_cpu_zo, &zo_outdatedOnCpu,
                 &d_gpu_zo, &zo_outdatedOnGpu, cpu_yc2, &yc2_outdatedOnCpu,
                 &c_gpu_yc2, &a__2_outdatedOnGpu, c_cpu_zo, &b_zo_outdatedOnCpu,
                 &e_gpu_zo, &b_zo_outdatedOnGpu);
        numStage = static_cast<real_T>(cpu_xc1->size[0]) - Letr;
        if (numStage > static_cast<real_T>(cpu_xc1->size[0]) - 1.0) {
          loop_ub = 0;
          i8 = 1;
          i10 = -1;
        } else {
          loop_ub = cpu_xc1->size[0] - 2;
          i8 = -1;
          i10 = static_cast<int32_T>(numStage) - 1;
        }
        i = r1->size[0] * r1->size[1];
        r1->size[0] = 1;
        r1->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(r1, i, &f_emlrtRTEI);
        i = b_xc1->size[0] * b_xc1->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        b_xc1->size[0] = div_s32(i10 - loop_ub, i8) + 1;
        b_xc1->size[1] = cpu_xc1->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_xc1, i, &db_emlrtRTEI);
        profileLoopStart("filtfilt_loop_36", __LINE__,
                         (cpu_xc1->size[1] - 1) + 1, "1724");
        for (i = 0; i < cpu_xc1->size[1]; i++) {
          if (a__4_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_real_T(cpu_xc1, &gpu_xc1);
          }
          a__4_outdatedOnGpu = false;
          r1->data[i] =
              2.0 *
              cpu_xc1->data[(cpu_xc1->size[0] + cpu_xc1->size[0] * i) - 1];
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_37", __LINE__,
                           div_s32(i10 - loop_ub, i8) + 1, "1725");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i10 - loop_ub, i8); b_i1++) {
            b_xc1->data[b_i1 + b_xc1->size[0] * i] =
                cpu_xc1->data[(loop_ub + i8 * b_i1) + cpu_xc1->size[0] * i];
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = g_cpu_B->size[0] * g_cpu_B->size[1];
        g_cpu_B->size[0] = 1;
        g_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(g_cpu_B, i, &fb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_40", __LINE__, (B->size[1] - 1) + 1,
                         "1726");
        for (i = 0; i < B->size[1]; i++) {
          g_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = g_cpu_A->size[0] * g_cpu_A->size[1];
        g_cpu_A->size[0] = 1;
        g_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(g_cpu_A, i, &gb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_43", __LINE__, (A->size[1] - 1) + 1,
                         "1727");
        for (i = 0; i < A->size[1]; i++) {
          g_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        if (r2_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_r2, &gpu_r2);
        }
        nvtxMarkA("#bsxfun#" MW_AT_LINE);
        bsxfun(r1, b_xc1, cpu_r2);
        r2_outdatedOnCpu = false;
        a__2_outdatedOnGpu = true;
        nvtxMarkA("#d_filter#" MW_AT_LINE);
        d_filter(g_cpu_B, &B_outdatedOnCpu, &j_gpu_B, &B_outdatedOnGpu, g_cpu_A,
                 &A_outdatedOnCpu, &j_gpu_A, &A_outdatedOnGpu, cpu_r2, &gpu_r2,
                 &a__2_outdatedOnGpu, c_cpu_zo, &b_zo_outdatedOnCpu, &e_gpu_zo,
                 &b_zo_outdatedOnGpu, cpu_yc3, &yc5_outdatedOnGpu, &c_gpu_yc3,
                 &yc3_outdatedOnGpu);
        if (cpu_yc3->size[0] < 1) {
          loop_ub = 0;
          i8 = 1;
          i10 = -1;
        } else {
          loop_ub = cpu_yc3->size[0] - 1;
          i8 = -1;
          i10 = 0;
        }
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = i_cpu_B->size[0] * i_cpu_B->size[1];
        i_cpu_B->size[0] = 1;
        i_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(i_cpu_B, i, &jb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_47", __LINE__, (B->size[1] - 1) + 1,
                         "1728");
        for (i = 0; i < B->size[1]; i++) {
          i_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = i_cpu_A->size[0] * i_cpu_A->size[1];
        i_cpu_A->size[0] = 1;
        i_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(i_cpu_A, i, &lb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_49", __LINE__, (A->size[1] - 1) + 1,
                         "1729");
        for (i = 0; i < A->size[1]; i++) {
          i_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        yc5_outdatedOnCpu = false;
        yc3_outdatedOnGpu = false;
        i = c_cpu_yc3->size[0] * c_cpu_yc3->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        c_cpu_yc3->size[0] = div_s32(i10 - loop_ub, i8) + 1;
        c_cpu_yc3->size[1] = cpu_yc3->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_cpu_yc3, i, &nb_emlrtRTEI);
        zi_outdatedOnCpu = false;
        zi_outdatedOnGpu = false;
        i = e_cpu_zi->size[0] * e_cpu_zi->size[1];
        e_cpu_zi->size[0] = zi->size[0];
        e_cpu_zi->size[1] = cpu_yc3->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(e_cpu_zi, i, &ob_emlrtRTEI);
        profileLoopStart("filtfilt_loop_51", __LINE__,
                         (cpu_yc3->size[1] - 1) + 1, "1730");
        for (i = 0; i < cpu_yc3->size[1]; i++) {
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_53", __LINE__,
                           div_s32(i10 - loop_ub, i8) + 1, "1731");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i10 - loop_ub, i8); b_i1++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(cpu_yc3, &c_gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            c_cpu_yc3->data[b_i1 + c_cpu_yc3->size[0] * i] =
                cpu_yc3->data[(loop_ub + i8 * b_i1) + cpu_yc3->size[0] * i];
            yc3_outdatedOnGpu = true;
          }
          profileLoopEnd();
          b_loop_ub = zi->size[0];
          profileLoopStart("filtfilt_loop_54", __LINE__, (b_loop_ub - 1) + 1,
                           "1732");
          for (b_i1 = 0; b_i1 < b_loop_ub; b_i1++) {
            if (yc5_outdatedOnGpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(cpu_yc3, &c_gpu_yc3);
            }
            yc5_outdatedOnGpu = false;
            e_cpu_zi->data[b_i1 + e_cpu_zi->size[0] * i] =
                zi->data[b_i1 + zi->size[0] * ii] *
                cpu_yc3->data[(cpu_yc3->size[0] + cpu_yc3->size[0] * i) - 1];
            zi_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        nvtxMarkA("#c_filter#" MW_AT_LINE);
        c_filter(i_cpu_B, &B_outdatedOnCpu, &k_gpu_B, &B_outdatedOnGpu, i_cpu_A,
                 &A_outdatedOnCpu, &k_gpu_A, &A_outdatedOnGpu, c_cpu_yc3,
                 &d_gpu_yc3, &yc3_outdatedOnGpu, e_cpu_zi, &zi_outdatedOnCpu,
                 &f_gpu_zi, &zi_outdatedOnGpu, cpu__4, &yc5_outdatedOnGpu,
                 &gpu__4, &a__4_outdatedOnGpu, e_cpu_zo, &b_zo_outdatedOnCpu,
                 &f_gpu_zo, &b_zo_outdatedOnGpu);
        if (cpu_yc2->size[0] < 1) {
          loop_ub = 0;
          i8 = 1;
          i10 = -1;
        } else {
          loop_ub = cpu_yc2->size[0] - 1;
          i8 = -1;
          i10 = 0;
        }
        B_outdatedOnCpu = false;
        B_outdatedOnGpu = false;
        i = k_cpu_B->size[0] * k_cpu_B->size[1];
        k_cpu_B->size[0] = 1;
        k_cpu_B->size[1] = B->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(k_cpu_B, i, &qb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_55", __LINE__, (B->size[1] - 1) + 1,
                         "1733");
        for (i = 0; i < B->size[1]; i++) {
          k_cpu_B->data[i] = B->data[ii + B->size[0] * i];
          B_outdatedOnGpu = true;
        }
        profileLoopEnd();
        A_outdatedOnCpu = false;
        A_outdatedOnGpu = false;
        i = k_cpu_A->size[0] * k_cpu_A->size[1];
        k_cpu_A->size[0] = 1;
        k_cpu_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(k_cpu_A, i, &sb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_57", __LINE__, (A->size[1] - 1) + 1,
                         "1734");
        for (i = 0; i < A->size[1]; i++) {
          k_cpu_A->data[i] = A->data[ii + A->size[0] * i];
          A_outdatedOnGpu = true;
        }
        profileLoopEnd();
        a__2_outdatedOnGpu = false;
        i = c_cpu_yc2->size[0] * c_cpu_yc2->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        c_cpu_yc2->size[0] = div_s32(i10 - loop_ub, i8) + 1;
        c_cpu_yc2->size[1] = cpu_yc2->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_cpu_yc2, i, &tb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_60", __LINE__,
                         (cpu_yc2->size[1] - 1) + 1, "1735");
        for (i = 0; i < cpu_yc2->size[1]; i++) {
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_61", __LINE__,
                           div_s32(i10 - loop_ub, i8) + 1, "1736");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i10 - loop_ub, i8); b_i1++) {
            if (yc2_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(cpu_yc2, &c_gpu_yc2);
            }
            yc2_outdatedOnCpu = false;
            c_cpu_yc2->data[b_i1 + c_cpu_yc2->size[0] * i] =
                cpu_yc2->data[(loop_ub + i8 * b_i1) + cpu_yc2->size[0] * i];
            a__2_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
        nvtxMarkA("#d_filter#" MW_AT_LINE);
        d_filter(k_cpu_B, &B_outdatedOnCpu, &l_gpu_B, &B_outdatedOnGpu, k_cpu_A,
                 &A_outdatedOnCpu, &l_gpu_A, &A_outdatedOnGpu, c_cpu_yc2,
                 &d_gpu_yc2, &a__2_outdatedOnGpu, e_cpu_zo, &b_zo_outdatedOnCpu,
                 &f_gpu_zo, &b_zo_outdatedOnGpu, cpu_yc5, &yc5_outdatedOnCpu,
                 &b_gpu_yc5, &yc5_outdatedOnGpu);
        if (cpu_yc5->size[0] < 1) {
          loop_ub = 0;
          i8 = 1;
          i10 = -1;
        } else {
          loop_ub = cpu_yc5->size[0] - 1;
          i8 = -1;
          i10 = 0;
        }
        a__4_outdatedOnGpu = false;
        yc3_outdatedOnGpu = false;
        i = cpu_xc1->size[0] * cpu_xc1->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        cpu_xc1->size[0] = div_s32(i10 - loop_ub, i8) + 1;
        cpu_xc1->size[1] = cpu_yc5->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_xc1, i, &vb_emlrtRTEI);
        profileLoopStart("filtfilt_loop_63", __LINE__,
                         (cpu_yc5->size[1] - 1) + 1, "1737");
        for (i = 0; i < cpu_yc5->size[1]; i++) {
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          profileLoopStart("filtfilt_loop_64", __LINE__,
                           div_s32(i10 - loop_ub, i8) + 1, "1738");
          nvtxMarkA("#div_s32#" MW_AT_LINE);
          for (b_i1 = 0; b_i1 <= div_s32(i10 - loop_ub, i8); b_i1++) {
            if (yc5_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real_T(cpu_yc5, &b_gpu_yc5);
            }
            yc5_outdatedOnCpu = false;
            cpu_xc1->data[b_i1 + cpu_xc1->size[0] * i] =
                cpu_yc5->data[(loop_ub + i8 * b_i1) + cpu_yc5->size[0] * i];
            yc3_outdatedOnGpu = true;
          }
          profileLoopEnd();
        }
        profileLoopEnd();
      }
      profileLoopEnd();
      i = y->size[0] * y->size[1] * y->size[2];
      y->size[0] = X->size[0];
      y->size[1] = X->size[1];
      y->size[2] = X->size[2];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(y, i, &g_emlrtRTEI);
      profileLoopStart(
          "filtfilt_loop_9", __LINE__,
          (static_cast<int32_T>(sz_idx_0) * static_cast<int32_T>(sz_idx_1) *
               static_cast<int32_T>(sz_idx_2) -
           1) +
              1,
          "1739");
      for (i = 0;
           i < static_cast<int32_T>(sz_idx_0) * static_cast<int32_T>(sz_idx_1) *
                   static_cast<int32_T>(sz_idx_2);
           i++) {
        y->data[i] = cpu_xc1->data[i];
      }
      profileLoopEnd();
    }
    if (xIsRow) {
      i = y->size[0] * y->size[1] * y->size[2];
      y->size[0] = 1;
      y->size[1] = X->size[0];
      y->size[2] = 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(y, i, &h_emlrtRTEI);
    }
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_ytemp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_r2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_yc2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&l_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&l_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_yc2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&k_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&k_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&f_cpu_zi);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_yc3);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&j_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&j_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&e_cpu_zi);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_yc3);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&i_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&i_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_xt);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&h_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&h_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&g_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&g_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_xc1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&r1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_zi);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&f_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&f_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&ytemp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_zi);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&e_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&e_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_zi);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_xt);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_zi);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&xc1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&r);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_yc5);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_yc5);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&f_cpu_zo);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu__2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&e_cpu_zo);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu__4);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_yc3);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_yc3);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_ytemp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_zo);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_yc2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_zo);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_yc2);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_zo);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu__3);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_zo);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu__1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xt);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&xt);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xc1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_ytemp);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&Y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&zi);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&B);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&X);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&den);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&num);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_ytemp);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xc1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_xt);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu__1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_zo);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu__3);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&e_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_ytemp);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu__4);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&f_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu__2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_zo);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_yc5);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_yc5);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&h_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&h_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&e_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&i_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&i_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xt);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_zi);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&j_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&j_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&e_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&e_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_xt);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&k_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&k_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&f_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&f_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&f_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_yc3);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_zi);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&l_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&l_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&g_gpu_B);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&g_gpu_A);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_yc2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_r2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_ytemp);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

} // namespace coder

// End of code generation (filtfilt.cu)

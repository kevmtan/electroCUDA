#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// uniformResampleKernel.cu
//
// Code generation for function 'uniformResampleKernel'
//

// Include files
#include "uniformResampleKernel.h"
#include "ec_cwt_data.h"
#include "ec_cwt_emxutil.h"
#include "ec_cwt_mexutil.h"
#include "ec_cwt_types.h"
#include "rt_nonfinite.h"
#include "upfirdn.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo yf_emlrtRTEI{
    59,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo ag_emlrtRTEI{
    17,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo bg_emlrtRTEI{
    23,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo cg_emlrtRTEI{
    55,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo dg_emlrtRTEI{
    13,                  // lineNo
    1,                   // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo eg_emlrtRTEI{
    14,                  // lineNo
    15,                  // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo fg_emlrtRTEI{
    16,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace b_signal {
namespace internal {
namespace resample {
void uniformResampleAlongFirstDim(
    emxArray_real32_T *cpu_xIn, real_T opts_p, real_T opts_q,
    const emxArray_real32_T *cpu_opts_filterWithPadding,
    real_T opts_filterDelay)
{
  emxArray_real32_T gpu_opts_filterWithPadding;
  emxArray_real32_T gpu_xIn;
  emxArray_real32_T gpu_yOut;
  emxArray_real32_T gpu_yRow;
  emxArray_real32_T *cpu_yOut;
  emxArray_real32_T *cpu_yRow;
  emxArray_real32_T *yRow;
  emxArray_real_T *b_y;
  emxArray_real_T *y;
  boolean_T c_opts_filterWithPadding_outdat;
  boolean_T d_opts_filterWithPadding_outdat;
  boolean_T xIn_outdatedOnCpu;
  boolean_T xIn_outdatedOnGpu;
  boolean_T yRow_outdatedOnCpu;
  boolean_T yRow_outdatedOnGpu;
  gpuEmxReset_real32_T(&gpu_yRow);
  gpuEmxReset_real32_T(&gpu_yOut);
  gpuEmxReset_real32_T(&gpu_opts_filterWithPadding);
  gpuEmxReset_real32_T(&gpu_xIn);
  yRow_outdatedOnCpu = false;
  yRow_outdatedOnGpu = false;
  c_opts_filterWithPadding_outdat = false;
  d_opts_filterWithPadding_outdat = true;
  xIn_outdatedOnCpu = false;
  xIn_outdatedOnGpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real32_T(&cpu_yOut, 2, &dg_emlrtRTEI, true);
  emxInit_real_T(&y, 2, &eg_emlrtRTEI, true);
  emxInit_real32_T(&cpu_yRow, 2, &fg_emlrtRTEI, true);
  emxInit_real_T(&b_y, 2, &w_emlrtRTEI, true);
  emxInit_real32_T(&yRow, 1, &bg_emlrtRTEI, true);
  if ((!(opts_p == 1.0)) || (!(opts_q == 1.0))) {
    if ((cpu_xIn->size[0] == 1) || (cpu_xIn->size[1] == 1)) {
      real_T b;
      int32_T b_i;
      int32_T i;
      int32_T u1;
      if ((cpu_xIn->size[0] == 0) || (cpu_xIn->size[1] == 0)) {
        u1 = 0;
      } else {
        i = cpu_xIn->size[0];
        u1 = cpu_xIn->size[1];
        if (i >= u1) {
          u1 = i;
        }
      }
      b = std::ceil(static_cast<real_T>(u1) * opts_p / opts_q);
      if ((cpu_xIn->size[0] == 1) && (cpu_xIn->size[1] == 1)) {
        upfirdn(cpu_xIn, &gpu_xIn, &xIn_outdatedOnGpu,
                (emxArray_real32_T *)cpu_opts_filterWithPadding,
                &c_opts_filterWithPadding_outdat, &gpu_opts_filterWithPadding,
                &d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
        if ((cpu_yRow->size[0] == 0) || (cpu_yRow->size[1] == 0)) {
          u1 = 0;
        } else {
          i = cpu_yRow->size[0];
          u1 = cpu_yRow->size[1];
          if (i >= u1) {
            u1 = i;
          }
        }
        b_i = cpu_yRow->size[0] * cpu_yRow->size[1];
        cpu_yRow->size[0] = u1;
        cpu_yRow->size[1] = 1;
        emxEnsureCapacity_real32_T(cpu_yRow, b_i, &ag_emlrtRTEI);
      } else {
        upfirdn(cpu_xIn, &gpu_xIn, &xIn_outdatedOnGpu,
                (emxArray_real32_T *)cpu_opts_filterWithPadding,
                &c_opts_filterWithPadding_outdat, &gpu_opts_filterWithPadding,
                &d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
      }
      if (std::isnan(b)) {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = 1;
        emxEnsureCapacity_real_T(b_y, b_i, &q_emlrtRTEI);
        b_y->data[0] = rtNaN;
      } else if (b < 1.0) {
        b_y->size[0] = 1;
        b_y->size[1] = 0;
      } else {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        emxEnsureCapacity_real_T(b_y, b_i, &q_emlrtRTEI);
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          b_y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
      }
      i = static_cast<int32_T>(b);
      b_i = yRow->size[0];
      yRow->size[0] = static_cast<int32_T>(b);
      emxEnsureCapacity_real32_T(yRow, b_i, &bg_emlrtRTEI);
      for (b_i = 0; b_i < i; b_i++) {
        if (yRow_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_yRow, &gpu_yRow);
        }
        yRow_outdatedOnCpu = false;
        yRow->data[b_i] =
            cpu_yRow
                ->data[static_cast<int32_T>(opts_filterDelay + b_y->data[b_i]) -
                       1];
      }
      i = static_cast<int32_T>(b);
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = static_cast<int32_T>(b);
      cpu_xIn->size[1] = 1;
      emxEnsureCapacity_real32_T(cpu_xIn, b_i, &cg_emlrtRTEI);
      for (u1 = 0; u1 < i; u1++) {
        if (xIn_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_xIn, &gpu_xIn);
        }
        cpu_xIn->data[u1] = yRow->data[u1];
        xIn_outdatedOnCpu = false;
      }
    } else {
      real_T b;
      int32_T b_i;
      int32_T i;
      b = std::ceil(static_cast<real_T>(cpu_xIn->size[0]) * opts_p / opts_q);
      upfirdn(cpu_xIn, &gpu_xIn, &xIn_outdatedOnGpu,
              (emxArray_real32_T *)cpu_opts_filterWithPadding,
              &c_opts_filterWithPadding_outdat, &gpu_opts_filterWithPadding,
              &d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yOut,
              &yRow_outdatedOnCpu, &gpu_yOut, &yRow_outdatedOnGpu);
      if (std::isnan(b)) {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = 1;
        emxEnsureCapacity_real_T(y, b_i, &q_emlrtRTEI);
        y->data[0] = rtNaN;
      } else if (b < 1.0) {
        y->size[0] = 1;
        y->size[1] = 0;
      } else {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        emxEnsureCapacity_real_T(y, b_i, &q_emlrtRTEI);
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
      }
      i = cpu_yOut->size[1];
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = y->size[1];
      cpu_xIn->size[1] = cpu_yOut->size[1];
      emxEnsureCapacity_real32_T(cpu_xIn, b_i, &yf_emlrtRTEI);
      for (b_i = 0; b_i < i; b_i++) {
        for (int32_T u1{0}; u1 < y->size[1]; u1++) {
          if (xIn_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_xIn, &gpu_xIn);
          }
          if (yRow_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_yOut, &gpu_yOut);
          }
          yRow_outdatedOnCpu = false;
          cpu_xIn->data[u1 + cpu_xIn->size[0] * b_i] =
              cpu_yOut
                  ->data[(static_cast<int32_T>(opts_filterDelay + y->data[u1]) +
                          cpu_yOut->size[0] * b_i) -
                         1];
          xIn_outdatedOnCpu = false;
        }
      }
    }
  }
  emxFree_real32_T(&yRow);
  emxFree_real_T(&b_y);
  emxFree_real32_T(&cpu_yRow);
  emxFree_real_T(&y);
  emxFree_real32_T(&cpu_yOut);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (xIn_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_xIn, &gpu_xIn);
  }
  gpuEmxFree_real32_T(&gpu_xIn);
  gpuEmxFree_real32_T(&gpu_opts_filterWithPadding);
  gpuEmxFree_real32_T(&gpu_yOut);
  gpuEmxFree_real32_T(&gpu_yRow);
}

} // namespace resample
} // namespace internal
} // namespace b_signal
} // namespace coder

// End of code generation (uniformResampleKernel.cu)

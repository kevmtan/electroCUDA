#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wt.cu
//
// Code generation for function 'wt'
//

// Include files
#include "wt.h"
#include "ec_cwt_data.h"
#include "ec_cwt_emxutil.h"
#include "ec_cwt_mexutil.h"
#include "ec_cwt_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hipfft/hipfft.h"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo xc_emlrtRTEI{
    102,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo yc_emlrtRTEI{
    135,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo ad_emlrtRTEI{
    143,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo bd_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "cuFFTNDCallback", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "cuFFTNDCallback.p" // pName
};

static emlrtRTEInfo cd_emlrtRTEI{
    138,  // lineNo
    19,   // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo dd_emlrtRTEI{
    138,  // lineNo
    6,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo ed_emlrtRTEI{
    161,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    180,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo gd_emlrtRTEI{
    151,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

// Function Declarations
static __global__ void cwtfilterbank_wt_kernel11(const int32_T i,
                                                 const int32_T b,
                                                 emxArray_real32_T x);

static __global__ void cwtfilterbank_wt_kernel12(const uint32_T xSize_dim1,
                                                 const int32_T b,
                                                 emxArray_creal32_T xposdft);

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel11(
    const int32_T i, const int32_T b, emxArray_real32_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    int32_T j2;
    real32_T xtmp;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel12(
    const uint32_T xSize_dim1, const int32_T b, emxArray_creal32_T xposdft)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    b_j1 = static_cast<int32_T>(idx);
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].re =
        xposdft.data[b_j1 + 1].re;
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].im =
        -xposdft.data[b_j1 + 1].im;
  }
}

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
}

//
//
namespace coder {
void cwtfilterbank_wt(const cwtfilterbank *self, const emxArray_real32_T *x,
                      emxArray_creal32_T *varargout_1)
{
  dim3 block;
  dim3 grid;
  emxArray_creal32_T gpu_cfsposdft;
  emxArray_creal32_T gpu_xposdft;
  emxArray_creal32_T *cpu_cfsposdft;
  emxArray_creal32_T *cpu_xposdft;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_xv;
  emxArray_real32_T *cpu_x;
  emxArray_real32_T *cpu_xv;
  emxArray_real32_T *psihat;
  real_T d;
  int32_T acoef;
  int32_T bcoef;
  int32_T csz_idx_1;
  int32_T i;
  int32_T i2;
  int32_T i3;
  int32_T nd2;
  uint32_T xSize[2];
  boolean_T validLaunchParams;
  boolean_T x_outdatedOnCpu;
  boolean_T xv_outdatedOnGpu;
  gpuEmxReset_creal32_T(&gpu_cfsposdft);
  gpuEmxReset_real32_T(&gpu_x);
  gpuEmxReset_creal32_T(&gpu_xposdft);
  gpuEmxReset_real32_T(&gpu_xv);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real32_T(&psihat, 2, &xc_emlrtRTEI, true);
  bcoef = psihat->size[0] * psihat->size[1];
  psihat->size[0] = self->PsiDFT->size[0];
  psihat->size[1] = self->PsiDFT->size[1];
  emxEnsureCapacity_real32_T(psihat, bcoef, &xc_emlrtRTEI);
  for (bcoef = 0; bcoef < self->PsiDFT->size[0] * self->PsiDFT->size[1];
       bcoef++) {
    psihat->data[bcoef] = static_cast<real32_T>(self->PsiDFT->data[bcoef]);
  }
  emxInit_real32_T(&cpu_xv, 2, &yc_emlrtRTEI, true);
  xv_outdatedOnGpu = false;
  bcoef = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = x->size[0];
  emxEnsureCapacity_real32_T(cpu_xv, bcoef, &yc_emlrtRTEI);
  for (bcoef = 0; bcoef < x->size[0]; bcoef++) {
    cpu_xv->data[bcoef] = x->data[bcoef];
    xv_outdatedOnGpu = true;
  }
  if (self->SignalPad > 0.0) {
    d = self->SignalPad;
    if (d < 1.0) {
      i = 0;
    } else {
      i = static_cast<int32_T>(d);
    }
    d = (static_cast<real_T>(x->size[0]) - self->SignalPad) + 1.0;
    if (d > x->size[0]) {
      csz_idx_1 = 0;
      i2 = 1;
      i3 = -1;
    } else {
      csz_idx_1 = x->size[0] - 1;
      i2 = -1;
      i3 = static_cast<int32_T>(d) - 1;
    }
    emxInit_real32_T(&cpu_x, 2, &cd_emlrtRTEI, true);
    xv_outdatedOnGpu = false;
    bcoef = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = 1;
    cpu_x->size[1] = i;
    emxEnsureCapacity_real32_T(cpu_x, bcoef, &cd_emlrtRTEI);
    for (bcoef = 0; bcoef < i; bcoef++) {
      cpu_x->data[bcoef] = x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    mwGetLaunchParameters1D(computeNumIters((i >> 1) - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !xv_outdatedOnGpu);
    if (xv_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      cwtfilterbank_wt_kernel11<<<grid, block>>>(i, (i >> 1) - 1, gpu_x);
    }
    x_outdatedOnCpu = true;
    xv_outdatedOnGpu = false;
    bcoef = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    cpu_xv->size[1] =
        ((x->size[0] + cpu_x->size[1]) + div_s32(i3 - csz_idx_1, i2)) + 1;
    emxEnsureCapacity_real32_T(cpu_xv, bcoef, &dd_emlrtRTEI);
    acoef = cpu_x->size[1];
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      if (x_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_x, &gpu_x);
      }
      x_outdatedOnCpu = false;
      cpu_xv->data[bcoef] = cpu_x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    acoef = x->size[0];
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      cpu_xv->data[bcoef + cpu_x->size[1]] = x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    acoef = div_s32(i3 - csz_idx_1, i2);
    for (bcoef = 0; bcoef <= acoef; bcoef++) {
      cpu_xv->data[(bcoef + x->size[0]) + cpu_x->size[1]] =
          x->data[csz_idx_1 + i2 * bcoef];
      xv_outdatedOnGpu = true;
    }
    emxFree_real32_T(&cpu_x);
  }
  emxInit_creal32_T(&cpu_xposdft, 2, &ad_emlrtRTEI, true);
  if (cpu_xv->size[1] == 0) {
    xv_outdatedOnGpu = false;
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = 0;
  } else if (cpu_xv->size[1] - (static_cast<int32_T>(
                                    static_cast<uint32_T>(cpu_xv->size[1]) >> 1)
                                << 1) ==
             1) {
    hipfftHandle b_fftPlanHandle;
    xv_outdatedOnGpu = false;
    bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = cpu_xv->size[1];
    emxEnsureCapacity_creal32_T(cpu_xposdft, bcoef, &ad_emlrtRTEI);
    for (bcoef = 0; bcoef < cpu_xv->size[1]; bcoef++) {
      cpu_xposdft->data[bcoef].re = cpu_xv->data[bcoef];
      cpu_xposdft->data[bcoef].im = 0.0F;
      xv_outdatedOnGpu = true;
    }
    nd2 = cpu_xposdft->size[1];
    b_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_C2C, 1);
    gpuEmxEnsureCapacity_creal32_T(cpu_xposdft, &gpu_xposdft,
                                   !xv_outdatedOnGpu);
    if (xv_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_xposdft, cpu_xposdft);
    }
    hipfftExecC2C(b_fftPlanHandle, (hipfftComplex *)&gpu_xposdft.data[0],
                 (hipfftComplex *)&gpu_xposdft.data[0], HIPFFT_FORWARD);
    xv_outdatedOnGpu = true;
  } else {
    hipfftHandle fftPlanHandle;
    nd2 = cpu_xv->size[1];
    xSize[1] = static_cast<uint32_T>(cpu_xv->size[1]);
    bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = cpu_xv->size[1];
    emxEnsureCapacity_creal32_T(cpu_xposdft, bcoef, &bd_emlrtRTEI);
    fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_R2C, 1);
    gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, !xv_outdatedOnGpu);
    gpuEmxEnsureCapacity_creal32_T(cpu_xposdft, &gpu_xposdft, true);
    if (xv_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_xv, cpu_xv);
    }
    hipfftExecR2C(fftPlanHandle, (hipfftReal *)&gpu_xv.data[0],
                 (hipfftComplex *)&gpu_xposdft.data[0]);
    nd2 = static_cast<int32_T>(static_cast<real_T>(cpu_xv->size[1] + 1) / 2.0);
    mwGetLaunchParameters1D(computeNumIters(nd2 - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      cwtfilterbank_wt_kernel12<<<grid, block>>>(xSize[1], nd2 - 2,
                                                 gpu_xposdft);
    }
    xv_outdatedOnGpu = true;
  }
  emxFree_real32_T(&cpu_xv);
  nd2 = psihat->size[1];
  i = cpu_xposdft->size[1];
  if (nd2 <= i) {
    i = nd2;
  }
  if (psihat->size[1] == 1) {
    csz_idx_1 = cpu_xposdft->size[1];
  } else if (cpu_xposdft->size[1] == 1) {
    csz_idx_1 = psihat->size[1];
  } else if (cpu_xposdft->size[1] == psihat->size[1]) {
    csz_idx_1 = cpu_xposdft->size[1];
  } else {
    csz_idx_1 = i;
  }
  emxInit_creal32_T(&cpu_cfsposdft, 2, &gd_emlrtRTEI, true);
  x_outdatedOnCpu = false;
  bcoef = cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
  cpu_cfsposdft->size[0] = psihat->size[0];
  nd2 = psihat->size[1];
  i = cpu_xposdft->size[1];
  if (nd2 <= i) {
    i = nd2;
  }
  if (psihat->size[1] == 1) {
    cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
  } else if (cpu_xposdft->size[1] == 1) {
    cpu_cfsposdft->size[1] = psihat->size[1];
  } else if (cpu_xposdft->size[1] == psihat->size[1]) {
    cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
  } else {
    cpu_cfsposdft->size[1] = i;
  }
  emxEnsureCapacity_creal32_T(cpu_cfsposdft, bcoef, &wc_emlrtRTEI);
  if ((psihat->size[0] != 0) && (csz_idx_1 != 0)) {
    acoef = (cpu_xposdft->size[1] != 1);
    bcoef = (psihat->size[1] != 1);
    csz_idx_1--;
    nd2 = (psihat->size[0] != 1);
    for (int32_T k{0}; k <= csz_idx_1; k++) {
      i = acoef * k;
      i2 = bcoef * k;
      i3 = cpu_cfsposdft->size[0] - 1;
      for (int32_T b_k{0}; b_k <= i3; b_k++) {
        if (xv_outdatedOnGpu) {
          gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xposdft, &gpu_xposdft);
        }
        cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].re =
            psihat->data[nd2 * b_k + psihat->size[0] * i2] *
            cpu_xposdft->data[i].re;
        xv_outdatedOnGpu = false;
        cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].im =
            psihat->data[nd2 * b_k + psihat->size[0] * i2] *
            cpu_xposdft->data[i].im;
        x_outdatedOnCpu = true;
      }
    }
  }
  emxFree_creal32_T(&cpu_xposdft);
  emxFree_real32_T(&psihat);
  if ((cpu_cfsposdft->size[0] == 0) || (cpu_cfsposdft->size[1] == 0)) {
    xSize[0] = static_cast<uint32_T>(cpu_cfsposdft->size[0]);
    xSize[1] = static_cast<uint32_T>(cpu_cfsposdft->size[1]);
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[0] = cpu_cfsposdft->size[0];
    varargout_1->size[1] = cpu_cfsposdft->size[1];
    emxEnsureCapacity_creal32_T(varargout_1, bcoef, &ed_emlrtRTEI);
    if (static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1 >=
        0) {
      std::memset(&varargout_1->data[0], 0,
                  static_cast<uint32_T>(static_cast<int32_T>(xSize[0]) *
                                        static_cast<int32_T>(xSize[1])) *
                      sizeof(creal32_T));
    }
  } else {
    hipfftHandle c_fftPlanHandle;
    real32_T cfsposdft_re;
    nd2 = cpu_cfsposdft->size[1];
    c_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, cpu_cfsposdft->size[0], 1,
                                       HIPFFT_C2C, cpu_cfsposdft->size[0]);
    gpuEmxEnsureCapacity_creal32_T(cpu_cfsposdft, &gpu_cfsposdft,
                                   !x_outdatedOnCpu);
    if (x_outdatedOnCpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_cfsposdft, cpu_cfsposdft);
    }
    hipfftExecC2C(c_fftPlanHandle, (hipfftComplex *)&gpu_cfsposdft.data[0],
                 (hipfftComplex *)&gpu_cfsposdft.data[0], HIPFFT_BACKWARD);
    xv_outdatedOnGpu = true;
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[0] = cpu_cfsposdft->size[0];
    varargout_1->size[1] = cpu_cfsposdft->size[1];
    emxEnsureCapacity_creal32_T(varargout_1, bcoef, &ed_emlrtRTEI);
    cfsposdft_re = static_cast<real32_T>(cpu_cfsposdft->size[1]);
    for (bcoef = 0; bcoef < cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
         bcoef++) {
      real32_T ai;
      real32_T ar;
      if (xv_outdatedOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal32_T(cpu_cfsposdft, &gpu_cfsposdft);
      }
      ar = cpu_cfsposdft->data[bcoef].re;
      xv_outdatedOnGpu = false;
      ai = cpu_cfsposdft->data[bcoef].im;
      if (ai == 0.0F) {
        varargout_1->data[bcoef].re = ar / cfsposdft_re;
        varargout_1->data[bcoef].im = 0.0F;
      } else if (ar == 0.0F) {
        varargout_1->data[bcoef].re = 0.0F;
        varargout_1->data[bcoef].im = ai / cfsposdft_re;
      } else {
        varargout_1->data[bcoef].re = ar / cfsposdft_re;
        varargout_1->data[bcoef].im = ai / cfsposdft_re;
      }
    }
  }
  emxFree_creal32_T(&cpu_cfsposdft);
  if (self->SignalPad > 0.0) {
    real_T d3;
    d = self->SignalPad + 1.0;
    d3 = self->SignalPad + self->SignalLength;
    if (d > d3) {
      nd2 = 0;
      i = 0;
    } else {
      nd2 = static_cast<int32_T>(d) - 1;
      i = static_cast<int32_T>(d3);
    }
    csz_idx_1 = varargout_1->size[0];
    acoef = i - nd2;
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      for (i2 = 0; i2 < csz_idx_1; i2++) {
        varargout_1->data[i2 + csz_idx_1 * bcoef] =
            varargout_1->data[i2 + varargout_1->size[0] * (nd2 + bcoef)];
      }
    }
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[1] = i - nd2;
    emxEnsureCapacity_creal32_T(varargout_1, bcoef, &fd_emlrtRTEI);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_xv);
  gpuEmxFree_creal32_T(&gpu_xposdft);
  gpuEmxFree_real32_T(&gpu_x);
  gpuEmxFree_creal32_T(&gpu_cfsposdft);
}

} // namespace coder

// End of code generation (wt.cu)

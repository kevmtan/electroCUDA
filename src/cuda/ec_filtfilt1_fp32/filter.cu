#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// filter.cu
//
// Code generation for function 'filter'
//

// Include files
#include "filter.h"
#include "ec_filtfilt1_data.h"
#include "ec_filtfilt1_emxutil.h"
#include "ec_filtfilt1_mexutil.h"
#include "ec_filtfilt1_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo l_emlrtRTEI{
    1,               // lineNo
    1,               // colNo
    "gpufilterImpl", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
    "gpufilterImpl.p" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "stencil_codegen", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "stencil_codegen.p" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    172,                                                          // lineNo
    20,                                                           // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    164,                                                               // lineNo
    9,                                                                 // colNo
    "filter",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/filter.m" // pName
};

// Function Declarations
static __global__ void filter_kernel20(const emxArray_real32_T a, real32_T *a1);

static __global__ void filter_kernel21(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel22(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T a);

static __global__ void filter_kernel23(emxArray_real32_T a);

static __global__ void filter_kernel24(const int32_T k, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel25(const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void filter_kernel26(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void filter_kernel27(const emxArray_real32_T x,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void
filter_kernel28(const emxArray_real32_T expanded, const emxArray_int32_T rows,
                const emxArray_real32_T b, const int32_T c,
                emxArray_real32_T convOut, int32_T b_dim0);

static __global__ void filter_kernel29(const emxArray_real32_T convOut,
                                       const int32_T na, const int32_T b,
                                       emxArray_real32_T zf);

static __global__ void filter_kernel30(const emxArray_real32_T convOut,
                                       const emxArray_real32_T a,
                                       const int32_T b, emxArray_real32_T zf,
                                       int32_T a_dim0, int32_T x_dim0);

static __global__ void filter_kernel31(const emxArray_real32_T convOut,
                                       const int32_T x, emxArray_real32_T y);

static __global__ void filter_kernel32(const emxArray_real32_T zi,
                                       const int32_T b,
                                       emxArray_real32_T convOut);

static __global__ void filter_kernel33(const emxArray_real32_T a, real32_T *a1);

static __global__ void filter_kernel34(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel35(const real32_T *a1, const int32_T b,
                                       emxArray_real32_T a);

static __global__ void filter_kernel36(emxArray_real32_T a);

static __global__ void filter_kernel37(const int32_T k, const int32_T b,
                                       emxArray_real32_T b_b);

static __global__ void filter_kernel38(const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void filter_kernel39(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void filter_kernel40(const emxArray_real32_T x,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_real32_T expanded);

static __global__ void
filter_kernel41(const emxArray_real32_T expanded, const emxArray_int32_T rows,
                const emxArray_real32_T b, const int32_T c,
                emxArray_real32_T convOut, int32_T b_dim0);

static __global__ void filter_kernel42(const emxArray_real32_T convOut,
                                       const int32_T x, emxArray_real32_T y);

static __global__ void filter_kernel43(const emxArray_real32_T zi,
                                       const int32_T b,
                                       emxArray_real32_T convOut);

static void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu,
                                         boolean_T needsCopy);

static void gpuEmxFree_int32_T(emxArray_int32_T *gpu);

static void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                         const emxArray_int32_T *cpu);

static void gpuEmxReset_int32_T(emxArray_int32_T *gpu);

// Function Definitions
static __global__
    __launch_bounds__(32, 1) void filter_kernel20(const emxArray_real32_T a,
                                                  real32_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel21(
    const real32_T *a1, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel22(const real32_T *a1,
                                                            const int32_T b,
                                                            emxArray_real32_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel23(emxArray_real32_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel24(
    const int32_T k, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    b_k = static_cast<int32_T>(idx);
    b_b.data[b_k + k] = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void filter_kernel25(const int32_T b,
                                                    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[k] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel26(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j;
    int32_T k;
    k = static_cast<int32_T>(idx);
    j = y.data[k];
    if ((offsetH < 0) && (j < MIN_int32_T - offsetH)) {
      j = MIN_int32_T;
    } else if ((offsetH > 0) && (j > MAX_int32_T - offsetH)) {
      j = MAX_int32_T;
    } else {
      j += offsetH;
    }
    iv.data[k] = j - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel27(
    const emxArray_real32_T x, const emxArray_int32_T iv, const int32_T b,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[iv.data[k]] = x.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel28(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_real32_T b, const int32_T c, emxArray_real32_T convOut,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    real32_T cv;
    k = static_cast<int32_T>(idx);
    cv = 0.0F;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T j;
      j = rows.data[m];
      if (j > 2147483646 - k) {
        j = MAX_int32_T;
      } else {
        j = (k + j) + 1;
      }
      cv += expanded.data[j - 1] * b.data[(b_dim0 - m) - 1];
    }
    convOut.data[k] = cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel29(
    const emxArray_real32_T convOut, const int32_T na, const int32_T b,
    emxArray_real32_T zf)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    zf.data[k] = convOut.data[na + k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel30(
    const emxArray_real32_T convOut, const emxArray_real32_T a, const int32_T b,
    emxArray_real32_T zf, int32_T a_dim0, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    i = static_cast<int32_T>(idx);
    k = static_cast<int32_T>(
        fmin(static_cast<real_T>(i) + 2.0, static_cast<real_T>(a_dim0)));
    for (int32_T j{0}; j <= k - 2; j++) {
      zf.data[i + 1] += convOut.data[(x_dim0 + i) - j] * a.data[j + 1];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel31(
    const emxArray_real32_T convOut, const int32_T x, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = convOut.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel32(
    const emxArray_real32_T zi, const int32_T b, emxArray_real32_T convOut)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    convOut.data[i] += zi.data[i];
  }
}

static __global__
    __launch_bounds__(32, 1) void filter_kernel33(const emxArray_real32_T a,
                                                  real32_T *a1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *a1 = a.data[0];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel34(
    const real32_T *a1, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_b.data[k] /= *a1;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void filter_kernel35(const real32_T *a1,
                                                            const int32_T b,
                                                            emxArray_real32_T a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    a.data[k + 1] /= *a1;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void filter_kernel36(emxArray_real32_T a)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a.data[0] = 1.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel37(
    const int32_T k, const int32_T b, emxArray_real32_T b_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    b_k = static_cast<int32_T>(idx);
    b_b.data[b_k + k] = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void filter_kernel38(const int32_T b,
                                                    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[k] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel39(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    int32_T q1;
    k = static_cast<int32_T>(idx);
    q1 = y.data[k];
    if ((offsetH < 0) && (q1 < MIN_int32_T - offsetH)) {
      q1 = MIN_int32_T;
    } else if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[k] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel40(
    const emxArray_real32_T x, const emxArray_int32_T iv, const int32_T b,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    expanded.data[iv.data[k]] = x.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel41(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_real32_T b, const int32_T c, emxArray_real32_T convOut,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    real32_T cv;
    k = static_cast<int32_T>(idx);
    cv = 0.0F;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - k) {
        q1 = MAX_int32_T;
      } else {
        q1 = (k + q1) + 1;
      }
      cv += expanded.data[q1 - 1] * b.data[(b_dim0 - m) - 1];
    }
    convOut.data[k] = cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel42(
    const emxArray_real32_T convOut, const int32_T x, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = convOut.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void filter_kernel43(
    const emxArray_real32_T zi, const int32_T b, emxArray_real32_T convOut)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    convOut.data[k] += zi.data[k];
  }
}

static void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu,
                                         boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  int32_T *newData;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(int32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(int32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxFree_int32_T(emxArray_int32_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                         const emxArray_int32_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(int32_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
}

static void gpuEmxReset_int32_T(emxArray_int32_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_int32_T));
}

//
//
namespace coder {
void b_filter(emxArray_real32_T *cpu_b, boolean_T *b_outdatedOnCpu,
              emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
              emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
              emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
              emxArray_real32_T *cpu_x, emxArray_real32_T *gpu_x,
              boolean_T *x_outdatedOnGpu, emxArray_real32_T *cpu_zi,
              boolean_T *zi_outdatedOnCpu, emxArray_real32_T *gpu_zi,
              boolean_T *zi_outdatedOnGpu, emxArray_real32_T *cpu_y,
              boolean_T *y_outdatedOnCpu, emxArray_real32_T *gpu_y,
              boolean_T *y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_real32_T gpu_convOut;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *cpu_convOut;
  emxArray_real32_T *cpu_expanded;
  int32_T k;
  int32_T na;
  int32_T nb;
  real32_T *gpu_a1;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  gpuEmxReset_real32_T(&gpu_convOut);
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&gpu_expanded);
  checkCudaError(mwCudaMalloc(&gpu_a1, 4UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[0];
  nb = cpu_b->size[0];
  gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
  }
  filter_kernel33<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  if (*a_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0])) &&
      (!(cpu_a->data[0] == 0.0F)) && (cpu_a->data[0] != 1.0F)) {
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel34<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
    mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel35<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
    }
    filter_kernel36<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
    *a_outdatedOnCpu = true;
  }
  if (cpu_a->size[0] > cpu_b->size[0]) {
    nb = cpu_a->size[0] - cpu_b->size[0];
    k = cpu_b->size[0];
    na = cpu_b->size[0];
    cpu_b->size[0] += nb;
    emxEnsureCapacity_real32_T(cpu_b, na, &l_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel37<<<grid, block>>>(k, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
  }
  if (cpu_x->size[0] == 0) {
    *y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
  } else {
    real_T blockDims_idx_0;
    int32_T KH;
    int32_T offsetH;
    uint32_T OH;
    uint32_T window_idx_0;
    int8_T threadDims_idx_0;
    boolean_T convOut_outdatedOnCpu;
    boolean_T convOut_outdatedOnGpu;
    window_idx_0 = static_cast<uint32_T>(cpu_b->size[0]);
    OH = (static_cast<uint32_T>(cpu_x->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    na = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    blockDims_idx_0 =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(na) - 1.0)) /
                   static_cast<real_T>(na));
    threadDims_idx_0 = static_cast<int8_T>(na);
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    KH = cpu_b->size[0];
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    emxInit_real32_T(&cpu_expanded, 1, &m_emlrtRTEI, true);
    if ((static_cast<int32_T>(OH) > 0) &&
        (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH))) {
      na = MAX_int32_T;
    } else {
      na = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0];
    cpu_expanded->size[0] = na - 1;
    emxEnsureCapacity_real32_T(cpu_expanded, k, &m_emlrtRTEI);
    na = (static_cast<int32_T>(OH) + static_cast<int32_T>(window_idx_0)) - 2;
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel38<<<grid, block>>>(na, gpu_expanded);
    }
    nb = cpu_x->size[0];
    emxInit_int32_T(&b_cpu_y, 2, &m_emlrtRTEI, true);
    k = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_x->size[0];
    emxEnsureCapacity_int32_T(b_cpu_y, k, &n_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    na = 1;
    for (k = 0; k <= nb - 2; k++) {
      na++;
      b_cpu_y->data[k + 1] = na;
    }
    emxInit_int32_T(&cpu_iv, 1, &m_emlrtRTEI, true);
    k = cpu_iv->size[0];
    cpu_iv->size[0] = b_cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv, k, &m_emlrtRTEI);
    na = b_cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel39<<<grid, block>>>(offsetH, b_gpu_y, na, gpu_iv);
    }
    emxFree_int32_T(&b_cpu_y);
    nb = cpu_iv->size[0];
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    if (*x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    *x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel40<<<grid, block>>>(*gpu_x, gpu_iv, nb - 1, gpu_expanded);
    }
    emxFree_int32_T(&cpu_iv);
    emxInit_int32_T(&cpu_rows, 2, &m_emlrtRTEI, true);
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = static_cast<int32_T>(window_idx_0);
    emxEnsureCapacity_int32_T(cpu_rows, k, &n_emlrtRTEI);
    cpu_rows->data[0] = 0;
    na = 0;
    for (k = 0; k <= KH - 2; k++) {
      na++;
      cpu_rows->data[k + 1] = na;
    }
    emxInit_real32_T(&cpu_convOut, 1, &l_emlrtRTEI, true);
    k = cpu_convOut->size[0];
    cpu_convOut->size[0] = static_cast<int32_T>(OH);
    emxEnsureCapacity_real32_T(cpu_convOut, k, &m_emlrtRTEI);
    if (blockDims_idx_0 < 4.294967296E+9) {
      if (blockDims_idx_0 >= 0.0) {
        window_idx_0 = static_cast<uint32_T>(blockDims_idx_0);
      } else {
        window_idx_0 = 0U;
      }
    } else if (blockDims_idx_0 >= 4.294967296E+9) {
      window_idx_0 = MAX_uint32_T;
    } else {
      window_idx_0 = 0U;
    }
    mwApplyLaunchParameters(
        computeNumIters(static_cast<int32_T>(OH) - 1),
        dim3(window_idx_0, 1U, 1U),
        dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid, &block);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    if (b_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel41<<<grid, block>>>(gpu_expanded, gpu_rows, *gpu_b,
                                       static_cast<int32_T>(OH) - 1,
                                       gpu_convOut, cpu_b->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    convOut_outdatedOnCpu = true;
    emxFree_int32_T(&cpu_rows);
    emxFree_real32_T(&cpu_expanded);
    if (cpu_zi->size[0] != 0) {
      if (cpu_zi->size[0] == 1) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        cpu_convOut->data[0] += cpu_zi->data[0];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      } else {
        na = cpu_zi->size[0];
        mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real32_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          filter_kernel43<<<grid, block>>>(*gpu_zi, na - 1, gpu_convOut);
        }
      }
    }
    na = cpu_x->size[0] + cpu_a->size[0];
    for (k = 0; k <= na - 2; k++) {
      nb = static_cast<int32_T>(std::fmin(static_cast<real_T>(k) + 1.0,
                                          static_cast<real_T>(cpu_a->size[0])));
      for (offsetH = 0; offsetH <= nb - 2; offsetH++) {
        if (convOut_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        }
        if (*a_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
        }
        *a_outdatedOnCpu = false;
        cpu_convOut->data[k] -=
            cpu_convOut->data[(k - offsetH) - 1] * cpu_a->data[offsetH + 1];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      }
    }
    k = cpu_y->size[0];
    cpu_y->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real32_T(cpu_y, k, &o_emlrtRTEI);
    na = cpu_x->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut,
                                    !convOut_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real32_T(cpu_y, gpu_y, true);
    if (convOut_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel42<<<grid, block>>>(gpu_convOut, na, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    emxFree_real32_T(&cpu_convOut);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  gpuEmxFree_real32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_rows);
  gpuEmxFree_real32_T(&gpu_convOut);
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
}

//
//
void filter(emxArray_real32_T *cpu_b, boolean_T *b_outdatedOnCpu,
            emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
            emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
            emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
            emxArray_real32_T *cpu_x, emxArray_real32_T *gpu_x,
            boolean_T *x_outdatedOnGpu, emxArray_real32_T *cpu_zi,
            boolean_T *zi_outdatedOnCpu, emxArray_real32_T *gpu_zi,
            boolean_T *zi_outdatedOnGpu, emxArray_real32_T *cpu_y,
            boolean_T *y_outdatedOnCpu, emxArray_real32_T *gpu_y,
            boolean_T *y_outdatedOnGpu, emxArray_real32_T *cpu_zf,
            boolean_T *zf_outdatedOnCpu, emxArray_real32_T *gpu_zf,
            boolean_T *zf_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_real32_T gpu_convOut;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *cpu_convOut;
  emxArray_real32_T *cpu_expanded;
  int32_T k;
  int32_T na;
  int32_T nb;
  int32_T zfSize_idx_0;
  real32_T *gpu_a1;
  boolean_T a_needsGpuEnsureCapacity;
  boolean_T b_needsGpuEnsureCapacity;
  boolean_T validLaunchParams;
  gpuEmxReset_real32_T(&gpu_convOut);
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&gpu_expanded);
  checkCudaError(mwCudaMalloc(&gpu_a1, 4UL), __FILE__, __LINE__);
  b_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  na = cpu_a->size[0];
  nb = cpu_b->size[0];
  gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
  if (*a_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
  }
  filter_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a, gpu_a1);
  if (*a_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
  }
  *a_outdatedOnCpu = false;
  if ((!std::isinf(cpu_a->data[0])) && (!std::isnan(cpu_a->data[0]))) {
    if (!(cpu_a->data[0] == 0.0F)) {
      if (cpu_a->data[0] != 1.0F) {
        mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
        b_needsGpuEnsureCapacity = false;
        if (*b_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          filter_kernel21<<<grid, block>>>(gpu_a1, nb - 1, *gpu_b);
        }
        *b_outdatedOnGpu = false;
        *b_outdatedOnCpu = true;
        mwGetLaunchParameters1D(computeNumIters(na - 2), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          filter_kernel22<<<grid, block>>>(gpu_a1, na - 2, *gpu_a);
        }
        a_needsGpuEnsureCapacity = false;
        filter_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_a);
        *a_outdatedOnCpu = true;
      } else {
        a_needsGpuEnsureCapacity = true;
      }
    } else {
      a_needsGpuEnsureCapacity = true;
    }
  } else {
    a_needsGpuEnsureCapacity = true;
  }
  if (cpu_a->size[0] > cpu_b->size[0]) {
    nb = cpu_a->size[0] - cpu_b->size[0];
    k = cpu_b->size[0];
    na = cpu_b->size[0];
    cpu_b->size[0] += nb;
    emxEnsureCapacity_real32_T(cpu_b, na, &l_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    b_needsGpuEnsureCapacity = false;
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel24<<<grid, block>>>(k, nb - 1, *gpu_b);
    }
    *b_outdatedOnGpu = false;
    *b_outdatedOnCpu = true;
  }
  zfSize_idx_0 = cpu_b->size[0] - 1;
  if (cpu_x->size[0] == 0) {
    *y_outdatedOnCpu = false;
    cpu_y->size[0] = 0;
    *zf_outdatedOnCpu = false;
    k = cpu_zf->size[0];
    cpu_zf->size[0] = cpu_b->size[0] - 1;
    emxEnsureCapacity_real32_T(cpu_zf, k, &l_emlrtRTEI);
  } else {
    real_T blockDims_idx_0;
    int32_T KH;
    int32_T offsetH;
    uint32_T OH;
    uint32_T window_idx_0;
    int8_T threadDims_idx_0;
    boolean_T convOut_outdatedOnCpu;
    boolean_T convOut_outdatedOnGpu;
    window_idx_0 = static_cast<uint32_T>(cpu_b->size[0]);
    OH = (static_cast<uint32_T>(cpu_x->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    na = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    blockDims_idx_0 =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(na) - 1.0)) /
                   static_cast<real_T>(na));
    threadDims_idx_0 = static_cast<int8_T>(na);
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    KH = cpu_b->size[0];
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    emxInit_real32_T(&cpu_expanded, 1, &m_emlrtRTEI, true);
    if ((static_cast<int32_T>(OH) > 0) &&
        (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH))) {
      na = MAX_int32_T;
    } else {
      na = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0];
    cpu_expanded->size[0] = na - 1;
    emxEnsureCapacity_real32_T(cpu_expanded, k, &m_emlrtRTEI);
    nb = (static_cast<int32_T>(OH) + static_cast<int32_T>(window_idx_0)) - 2;
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel25<<<grid, block>>>(nb, gpu_expanded);
    }
    nb = cpu_x->size[0];
    emxInit_int32_T(&b_cpu_y, 2, &m_emlrtRTEI, true);
    k = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_x->size[0];
    emxEnsureCapacity_int32_T(b_cpu_y, k, &n_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    na = 1;
    for (k = 0; k <= nb - 2; k++) {
      na++;
      b_cpu_y->data[k + 1] = na;
    }
    emxInit_int32_T(&cpu_iv, 1, &m_emlrtRTEI, true);
    k = cpu_iv->size[0];
    cpu_iv->size[0] = b_cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv, k, &m_emlrtRTEI);
    nb = b_cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel26<<<grid, block>>>(offsetH, b_gpu_y, nb, gpu_iv);
    }
    emxFree_int32_T(&b_cpu_y);
    nb = cpu_iv->size[0];
    mwGetLaunchParameters1D(computeNumIters(nb - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    if (*x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel27<<<grid, block>>>(*gpu_x, gpu_iv, nb - 1, gpu_expanded);
    }
    emxFree_int32_T(&cpu_iv);
    emxInit_int32_T(&cpu_rows, 2, &m_emlrtRTEI, true);
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = static_cast<int32_T>(window_idx_0);
    emxEnsureCapacity_int32_T(cpu_rows, k, &n_emlrtRTEI);
    cpu_rows->data[0] = 0;
    na = 0;
    for (k = 0; k <= KH - 2; k++) {
      na++;
      cpu_rows->data[k + 1] = na;
    }
    emxInit_real32_T(&cpu_convOut, 1, &l_emlrtRTEI, true);
    k = cpu_convOut->size[0];
    cpu_convOut->size[0] = static_cast<int32_T>(OH);
    emxEnsureCapacity_real32_T(cpu_convOut, k, &m_emlrtRTEI);
    if (blockDims_idx_0 < 4.294967296E+9) {
      if (blockDims_idx_0 >= 0.0) {
        window_idx_0 = static_cast<uint32_T>(blockDims_idx_0);
      } else {
        window_idx_0 = 0U;
      }
    } else if (blockDims_idx_0 >= 4.294967296E+9) {
      window_idx_0 = MAX_uint32_T;
    } else {
      window_idx_0 = 0U;
    }
    mwApplyLaunchParameters(
        computeNumIters(static_cast<int32_T>(OH) - 1),
        dim3(window_idx_0, 1U, 1U),
        dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid, &block);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    if (b_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut, true);
    b_needsGpuEnsureCapacity = false;
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel28<<<grid, block>>>(gpu_expanded, gpu_rows, *gpu_b,
                                       static_cast<int32_T>(OH) - 1,
                                       gpu_convOut, cpu_b->size[0U]);
    }
    convOut_outdatedOnGpu = false;
    convOut_outdatedOnCpu = true;
    emxFree_int32_T(&cpu_rows);
    emxFree_real32_T(&cpu_expanded);
    if (cpu_zi->size[0] != 0) {
      if (cpu_zi->size[0] == 1) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        if (*zi_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_zi, gpu_zi);
        }
        *zi_outdatedOnCpu = false;
        cpu_convOut->data[0] += cpu_zi->data[0];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        b_needsGpuEnsureCapacity = true;
      } else {
        na = cpu_zi->size[0];
        mwGetLaunchParameters1D(computeNumIters(na - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real32_T(cpu_zi, gpu_zi, !*zi_outdatedOnGpu);
        if (*zi_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_zi, cpu_zi);
        }
        *zi_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          filter_kernel32<<<grid, block>>>(*gpu_zi, na - 1, gpu_convOut);
        }
      }
    }
    na = cpu_x->size[0] + cpu_a->size[0];
    for (k = 0; k <= na - 2; k++) {
      nb = static_cast<int32_T>(std::fmin(static_cast<real_T>(k) + 1.0,
                                          static_cast<real_T>(cpu_a->size[0])));
      for (offsetH = 0; offsetH <= nb - 2; offsetH++) {
        if (convOut_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_convOut, &gpu_convOut);
        }
        if (*a_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
        }
        *a_outdatedOnCpu = false;
        cpu_convOut->data[k] -=
            cpu_convOut->data[(k - offsetH) - 1] * cpu_a->data[offsetH + 1];
        convOut_outdatedOnCpu = false;
        convOut_outdatedOnGpu = true;
        a_needsGpuEnsureCapacity = true;
        b_needsGpuEnsureCapacity = true;
      }
    }
    if (static_cast<uint32_T>(cpu_x->size[0]) + 1U >
        static_cast<uint32_T>(cpu_convOut->size[0])) {
      na = 0;
      nb = 0;
    } else {
      na = cpu_x->size[0];
      nb = cpu_convOut->size[0];
    }
    k = cpu_zf->size[0];
    cpu_zf->size[0] = nb - na;
    emxEnsureCapacity_real32_T(cpu_zf, k, &l_emlrtRTEI);
    nb = (nb - na) - 1;
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    if (b_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_convOut, &gpu_convOut,
                                    !convOut_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real32_T(cpu_zf, gpu_zf, true);
    if (convOut_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_convOut, cpu_convOut);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel29<<<grid, block>>>(gpu_convOut, na, nb, *gpu_zf);
    }
    na = cpu_a->size[0];
    mwGetLaunchParameters1D(computeNumIters(na - 3), &grid, &block,
                            2147483647U);
    if (a_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, true);
    }
    *x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel30<<<grid, block>>>(gpu_convOut, *gpu_a, na - 3, *gpu_zf,
                                       cpu_a->size[0U], cpu_x->size[0U]);
    }
    *zf_outdatedOnCpu = true;
    k = cpu_zf->size[0];
    cpu_zf->size[0] = zfSize_idx_0;
    emxEnsureCapacity_real32_T(cpu_zf, k, &l_emlrtRTEI);
    k = cpu_y->size[0];
    cpu_y->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real32_T(cpu_y, k, &l_emlrtRTEI);
    nb = cpu_x->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(nb), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_y, gpu_y, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      filter_kernel31<<<grid, block>>>(gpu_convOut, nb, *gpu_y);
    }
    *y_outdatedOnCpu = true;
    emxFree_real32_T(&cpu_convOut);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  checkCudaError(mwCudaFree(gpu_a1), __FILE__, __LINE__);
  gpuEmxFree_real32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_rows);
  gpuEmxFree_real32_T(&gpu_convOut);
  *a_outdatedOnGpu = false;
  *y_outdatedOnGpu = false;
  *zf_outdatedOnGpu = false;
}

} // namespace coder

// End of code generation (filter.cu)

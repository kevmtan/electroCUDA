//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_filtfilt1_initialize.cu
//
// Code generation for function 'ec_filtfilt1_initialize'
//

// Include files
#include "ec_filtfilt1_initialize.h"
#include "_coder_ec_filtfilt1_mex.h"
#include "ec_filtfilt1_data.h"
#include "rt_nonfinite.h"

// Function Definitions
void ec_filtfilt1_initialize()
{
  mex_InitInfAndNan();
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

// End of code generation (ec_filtfilt1_initialize.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_filtfilt1.cu
//
// Code generation for function 'ec_filtfilt1'
//

// Include files
#include "ec_filtfilt1.h"
#include "ec_filtfilt1_data.h"
#include "ec_filtfilt1_emxutil.h"
#include "ec_filtfilt1_mexutil.h"
#include "ec_filtfilt1_types.h"
#include "filter.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    99,                                                         // lineNo
    1,                                                          // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    103,                                                        // lineNo
    28,                                                         // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    60,                                                              // lineNo
    20,                                                              // colNo
    "bsxfun",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/bsxfun.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    103,                                                        // lineNo
    15,                                                         // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo
    e_emlrtRTEI{
        76,                  // lineNo
        13,                  // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo f_emlrtRTEI{
    103,                                                        // lineNo
    5,                                                          // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    132,                                                        // lineNo
    5,                                                          // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    1,                                                          // lineNo
    14,                                                         // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    135,                                                        // lineNo
    10,                                                         // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    136,                                                        // lineNo
    5,                                                          // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    139,                                                        // lineNo
    5,                                                          // colNo
    "ec_filtfilt1",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_filtfilt1.m" // pName
};

// Function Declarations
static int32_T div_s32(int32_T numerator, int32_T denominator);

static __global__ void ec_filtfilt1_kernel1(const int32_T ch,
                                            const emxArray_real_T x,
                                            const int32_T b_x,
                                            emxArray_real_T dv, int32_T x_dim0);

static __global__ void ec_filtfilt1_kernel10(const emxArray_real_T a,
                                             const int32_T b_a,
                                             emxArray_real_T c_a);

static __global__ void ec_filtfilt1_kernel11(const emxArray_real_T b,
                                             const int32_T b_b,
                                             emxArray_real_T c_b);

static __global__ void ec_filtfilt1_kernel12(const emxArray_real_T a,
                                             const int32_T b_a,
                                             emxArray_real_T c_a);

static __global__ void ec_filtfilt1_kernel13(const emxArray_real_T yc3,
                                             const int32_T bcoef,
                                             const int32_T i5, const int32_T b,
                                             emxArray_real_T b_yc3);

static __global__ void ec_filtfilt1_kernel14(const real_T a,
                                             const emxArray_real_T z,
                                             const int32_T b_z,
                                             emxArray_real_T c_z);

static __global__ void ec_filtfilt1_kernel15(const emxArray_real_T b,
                                             const int32_T b_b,
                                             emxArray_real_T c_b);

static __global__ void ec_filtfilt1_kernel16(const emxArray_real_T a,
                                             const int32_T b_a,
                                             emxArray_real_T c_a);

static __global__ void ec_filtfilt1_kernel17(const emxArray_real_T yc2,
                                             const int32_T bcoef,
                                             const int32_T i5, const int32_T b,
                                             emxArray_real_T b_yc2);

static __global__ void ec_filtfilt1_kernel18(const emxArray_real_T yc5,
                                             const int32_T bcoef,
                                             const int32_T i5, const int32_T b,
                                             emxArray_real_T dv);

static __global__ void ec_filtfilt1_kernel19(const emxArray_real_T dv,
                                             const int32_T ch, const int32_T b,
                                             emxArray_real_T y, int32_T y_dim0);

static __global__ void ec_filtfilt1_kernel2(const emxArray_real_T dv,
                                            const int32_T bcoef,
                                            const int32_T i3, const int32_T i2,
                                            const real_T a, const int32_T i5,
                                            emxArray_real_T xt);

static __global__ void ec_filtfilt1_kernel3(const emxArray_real_T b,
                                            const int32_T b_b,
                                            emxArray_real_T c_b);

static __global__ void ec_filtfilt1_kernel4(const emxArray_real_T a,
                                            const int32_T b_a,
                                            emxArray_real_T c_a);

static __global__ void ec_filtfilt1_kernel5(const real_T a,
                                            const emxArray_real_T z,
                                            const int32_T b_z,
                                            emxArray_real_T c_z);

static __global__ void ec_filtfilt1_kernel6(const emxArray_real_T b,
                                            const int32_T b_b,
                                            emxArray_real_T c_b);

static __global__ void ec_filtfilt1_kernel7(const emxArray_real_T a,
                                            const int32_T b_a,
                                            emxArray_real_T c_a);

static __global__ void ec_filtfilt1_kernel8(const emxArray_real_T dv,
                                            const int32_T bcoef,
                                            const int32_T i7, const int32_T i6,
                                            const real_T a, const int32_T i5,
                                            emxArray_real_T c);

static __global__ void ec_filtfilt1_kernel9(const emxArray_real_T b,
                                            const int32_T b_b,
                                            emxArray_real_T c_b);

// Function Definitions
static int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel1(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T dv, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    dv.data[k] = x.data[k + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel10(
    const emxArray_real_T a, const int32_T b_a, emxArray_real_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel11(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel12(
    const emxArray_real_T a, const int32_T b_a, emxArray_real_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel13(
    const emxArray_real_T yc3, const int32_T bcoef, const int32_T i5,
    const int32_T b, emxArray_real_T b_yc3)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_yc3.data[k] = yc3.data[i5 + bcoef * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel14(
    const real_T a, const emxArray_real_T z, const int32_T b_z,
    emxArray_real_T c_z)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_z);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_z.data[k] = z.data[k] * a;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel15(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel16(
    const emxArray_real_T a, const int32_T b_a, emxArray_real_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel17(
    const emxArray_real_T yc2, const int32_T bcoef, const int32_T i5,
    const int32_T b, emxArray_real_T b_yc2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_yc2.data[k] = yc2.data[i5 + bcoef * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel18(
    const emxArray_real_T yc5, const int32_T bcoef, const int32_T i5,
    const int32_T b, emxArray_real_T dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    dv.data[k] = yc5.data[i5 + bcoef * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel19(
    const emxArray_real_T dv, const int32_T ch, const int32_T b,
    emxArray_real_T y, int32_T y_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k + y_dim0 * ch] = dv.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel2(
    const emxArray_real_T dv, const int32_T bcoef, const int32_T i3,
    const int32_T i2, const real_T a, const int32_T i5, emxArray_real_T xt)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i5);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    xt.data[k] = a - dv.data[(i2 + i3 * (bcoef * k)) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel3(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel4(
    const emxArray_real_T a, const int32_T b_a, emxArray_real_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel5(
    const real_T a, const emxArray_real_T z, const int32_T b_z,
    emxArray_real_T c_z)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_z);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_z.data[k] = z.data[k] * a;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel6(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel7(
    const emxArray_real_T a, const int32_T b_a, emxArray_real_T c_a)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_a.data[k] = a.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel8(
    const emxArray_real_T dv, const int32_T bcoef, const int32_T i7,
    const int32_T i6, const real_T a, const int32_T i5, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(i5);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c.data[k] = a - dv.data[(i6 + i7 * (bcoef * k)) - 1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_filtfilt1_kernel9(
    const emxArray_real_T b, const int32_T b_b, emxArray_real_T c_b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_b.data[k] = b.data[k];
  }
}

//
// function y = ec_filtfilt1(x,b,a,z,nf,L)
void ec_filtfilt1(const emxArray_real_T *cpu_x, const emxArray_real_T *cpu_b,
                  const emxArray_real_T *cpu_a, const emxArray_real_T *cpu_z,
                  real_T nf, real_T L, emxArray_real_T *cpu_y)
{
  dim3 block;
  dim3 grid;
  emxArray_real_T b_gpu_a;
  emxArray_real_T b_gpu_b;
  emxArray_real_T b_gpu_yc2;
  emxArray_real_T b_gpu_yc3;
  emxArray_real_T b_gpu_z;
  emxArray_real_T b_gpu_zo;
  emxArray_real_T c_gpu_a;
  emxArray_real_T c_gpu_b;
  emxArray_real_T c_gpu_z;
  emxArray_real_T c_gpu_zo;
  emxArray_real_T d_gpu_a;
  emxArray_real_T d_gpu_b;
  emxArray_real_T e_gpu_a;
  emxArray_real_T e_gpu_b;
  emxArray_real_T f_gpu_a;
  emxArray_real_T f_gpu_b;
  emxArray_real_T gpu__1;
  emxArray_real_T gpu__2;
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_b;
  emxArray_real_T gpu_c;
  emxArray_real_T gpu_dv;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xt;
  emxArray_real_T gpu_y;
  emxArray_real_T gpu_yc2;
  emxArray_real_T gpu_yc3;
  emxArray_real_T gpu_yc5;
  emxArray_real_T gpu_z;
  emxArray_real_T gpu_zo;
  emxArray_real_T *b_cpu_a;
  emxArray_real_T *b_cpu_b;
  emxArray_real_T *b_cpu_yc2;
  emxArray_real_T *b_cpu_yc3;
  emxArray_real_T *b_cpu_z;
  emxArray_real_T *b_cpu_zo;
  emxArray_real_T *c_cpu_a;
  emxArray_real_T *c_cpu_b;
  emxArray_real_T *c_cpu_z;
  emxArray_real_T *c_cpu_zo;
  emxArray_real_T *cpu__1;
  emxArray_real_T *cpu__2;
  emxArray_real_T *cpu_c;
  emxArray_real_T *cpu_dv;
  emxArray_real_T *cpu_xt;
  emxArray_real_T *cpu_yc2;
  emxArray_real_T *cpu_yc3;
  emxArray_real_T *cpu_yc5;
  emxArray_real_T *cpu_zo;
  emxArray_real_T *d_cpu_a;
  emxArray_real_T *d_cpu_b;
  emxArray_real_T *e_cpu_a;
  emxArray_real_T *e_cpu_b;
  emxArray_real_T *f_cpu_a;
  emxArray_real_T *f_cpu_b;
  int32_T b_i;
  int32_T csz_idx_0;
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T i3;
  int32_T i4;
  boolean_T a_outdatedOnCpu;
  boolean_T a_outdatedOnGpu;
  boolean_T b_outdatedOnCpu;
  boolean_T b_outdatedOnGpu;
  boolean_T b_zo_outdatedOnGpu;
  boolean_T dv_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T xt_outdatedOnCpu;
  boolean_T y_needsGpuEnsureCapacity;
  boolean_T yc2_outdatedOnCpu;
  boolean_T yc2_outdatedOnGpu;
  boolean_T yc3_outdatedOnCpu;
  boolean_T yc3_outdatedOnGpu;
  boolean_T yc5_outdatedOnCpu;
  boolean_T yc5_outdatedOnGpu;
  boolean_T z_needsGpuEnsureCapacity;
  boolean_T z_outdatedOnGpu;
  boolean_T zo_outdatedOnCpu;
  boolean_T zo_outdatedOnGpu;
  gpuEmxReset_real_T(&b_gpu_yc2);
  gpuEmxReset_real_T(&f_gpu_a);
  gpuEmxReset_real_T(&f_gpu_b);
  gpuEmxReset_real_T(&c_gpu_z);
  gpuEmxReset_real_T(&b_gpu_yc3);
  gpuEmxReset_real_T(&e_gpu_a);
  gpuEmxReset_real_T(&e_gpu_b);
  gpuEmxReset_real_T(&d_gpu_a);
  gpuEmxReset_real_T(&d_gpu_b);
  gpuEmxReset_real_T(&c_gpu_a);
  gpuEmxReset_real_T(&c_gpu_b);
  gpuEmxReset_real_T(&b_gpu_z);
  gpuEmxReset_real_T(&b_gpu_a);
  gpuEmxReset_real_T(&b_gpu_b);
  gpuEmxReset_real_T(&gpu_yc5);
  gpuEmxReset_real_T(&c_gpu_zo);
  gpuEmxReset_real_T(&gpu__2);
  gpuEmxReset_real_T(&gpu_yc3);
  gpuEmxReset_real_T(&gpu_c);
  gpuEmxReset_real_T(&b_gpu_zo);
  gpuEmxReset_real_T(&gpu_yc2);
  gpuEmxReset_real_T(&gpu_zo);
  gpuEmxReset_real_T(&gpu__1);
  gpuEmxReset_real_T(&gpu_xt);
  gpuEmxReset_real_T(&gpu_dv);
  gpuEmxReset_real_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_z);
  gpuEmxReset_real_T(&gpu_a);
  gpuEmxReset_real_T(&gpu_b);
  gpuEmxReset_real_T(&gpu_x);
  zo_outdatedOnCpu = false;
  b_zo_outdatedOnGpu = false;
  yc2_outdatedOnGpu = false;
  yc5_outdatedOnGpu = false;
  z_outdatedOnGpu = true;
  a_outdatedOnGpu = true;
  b_outdatedOnGpu = true;
  x_outdatedOnGpu = true;
  z_needsGpuEnsureCapacity = true;
  yc3_outdatedOnGpu = true;
  zo_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  [ec_filtfilt1] helper function for ec_filtfilt()
  //  FILTFILT Zero-phase forward and reverse digital IIR filtering.
  //    Y = FILTFILT(B,A,X) filters the data in vector, matrix, or N-D
  //    array, X, with the filter described by vectors A and B to create
  //    the filtered data Y. The filter is described by the difference
  //    equation:
  //
  //      a(1)*y(n) = b(1)*x(n) + b(2)*x(n-1) + ... + b(nb+1)*x(n-nb)
  //                            - a(2)*y(n-1) - ... - a(na+1)*y(n-na)
  //
  //    filtfilt operates along the first dimension of X except when X is a
  //    row vector in which case it operates along the second dimension.
  //    The length of the input channels must be more than three times the
  //    filter order, defined as max(length(B)-1,length(A)-1).
  //
  //    Y = FILTFILT(SOS,G,X) filters the data in vector, matrix, or N-D
  //    array, X, with the second-order section (SOS) filter described by
  //    the matrix SOS and the vector G. The coefficients of the SOS matrix
  //    must be expressed using an Lx6 matrix where L is the number of
  //    second-order sections. The scale values of the filter must be
  //    expressed using the vector G. The length of G must be between 1 and
  //    L+1, and the length of input channels in X must be more than three
  //    times the filter order (input channel length must be greater than
  //    one when the order is zero). You can use filtord(SOS) to get the
  //    order of the filter. The SOS matrix should have the following form:
  //
  //    SOS = [ b01 b11 b21 a01 a11 a21
  //            b02 b12 b22 a02 a12 a22
  //            ...
  //            b0L b1L b2L a0L a1L a2L ]
  //
  //    Y = FILTFILT(D,X) filters the data in vector, matrix, or N-D
  //    array, X, with the digital filter D. You design a digital filter,
  //    D, by calling the <a href="matlab:help designfilt">designfilt</a>
  //    function. The length of the input channels in X must be more than
  //    three times the filter order. You can use filtord(D) to get the
  //    order of the digital filter D.
  //
  //    After filtering in the forward direction, the filtered data is
  //    reversed and run back through the filter; Y is the time reverse of
  //    the output of the second filtering operation. The result has
  //    precisely zero phase distortion, and magnitude modified by the
  //    square of the filter's magnitude response. Startup and ending
  //    transients are minimized by matching initial conditions.
  //
  //    Note that FILTFILT should not be used when the intent of a filter
  //    is to modify signal phase, such as differentiators and Hilbert
  //    filters.
  //
  //    % Example 1:
  //    %   Zero-phase filter a noisy ECG waveform using an IIR filter.
  //
  //    load noisysignals x;                    % noisy waveform
  //    [b,a] = butter(12,0.2,'low');           % IIR filter design
  //    y = filtfilt(b,a,x);                    % zero-phase filtering
  //    y2 = filter(b,a,x);                     % conventional filtering
  //    plot(x,'k-.'); grid on ; hold on
  //    plot([y y2],'LineWidth',1.5);
  //    legend('Noisy ECG','Zero-phase Filtering','Conventional Filtering');
  //
  //    % Example 2:
  //    %   Use the designfilt function to design a highpass IIR digital
  //    %   filter with order 4, passband frequency of 75 KHz, and a passband
  //    %   ripple of 0.2 dB. Sample rate is 200 KHz. Apply zero-phase
  //    %   filtering to a vector of data.
  //
  //    D = designfilt('highpassiir', 'FilterOrder', 4, ...
  //             'PassbandFrequency', 75e3, 'PassbandRipple', 0.2,...
  //             'SampleRate', 200e3);
  //
  //    x = rand(1000,1);
  //    y = filtfilt(D,x);
  //
  //    See also FILTER, SOSFILT.
  //    References:
  //      [1] Sanjit K. Mitra, Digital Signal Processing, 2nd ed.,
  //          McGraw-Hill, 2001
  //      [2] Fredrik Gustafsson, Determining the initial states in forward-
  //          backward filtering, IEEE Transactions on Signal Processing,
  //          pp. 988-992, April 1996, Volume 44, Issue 4
  //    Copyright 1988-2022 The MathWorks, Inc.
  //
  //  For filtfilt(D,X), the inputs passed to the function are B,A,X thanks
  //  to the method of D.
  //  Input validation
  // 'ec_filtfilt1:89' lfn = coder.target('MATLAB');
  // 'ec_filtfilt1:90' if ~lfn
  // 'ec_filtfilt1:91'
  // validateattributes(b,{'double','single'},{'2d','finite','nonempty'},'filtfilt');
  // 'ec_filtfilt1:92'
  // validateattributes(a,{'double','single'},{'2d','finite','vector','nonempty'},'filtfilt');
  // 'ec_filtfilt1:93'
  // validateattributes(x,{'double','single'},{'2d','finite','nonempty'},'filtfilt');
  //  Run
  // 'ec_filtfilt1:99' y = coder.nullcopy(x);
  yc5_outdatedOnCpu = false;
  i = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = cpu_x->size[0];
  cpu_y->size[1] = cpu_x->size[1];
  emxEnsureCapacity_real_T(cpu_y, i, &emlrtRTEI);
  y_needsGpuEnsureCapacity = true;
  //  Loop across chans
  // 'ec_filtfilt1:102' for ch = 1:width(x)
  b_i = cpu_x->size[1];
  if (cpu_x->size[1] - 1 >= 0) {
    i1 = static_cast<int32_T>(L);
    if (static_cast<int32_T>(L) - 1 >= 0) {
      if (nf + 1.0 < 2.0) {
        i2 = 1;
        i3 = 1;
        i4 = 0;
      } else {
        i2 = static_cast<int32_T>(nf + 1.0);
        i3 = -1;
        i4 = 2;
      }
      csz_idx_0 = div_s32(i4 - i2, i3) + 1;
    }
  }
  emxInit_real_T(&cpu_dv, 1, &f_emlrtRTEI, true);
  emxInit_real_T(&cpu_xt, 1, &g_emlrtRTEI, true);
  emxInit_real_T(&cpu__1, 1, &h_emlrtRTEI, true);
  emxInit_real_T(&cpu_zo, 1, &h_emlrtRTEI, true);
  emxInit_real_T(&cpu_yc2, 1, &h_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_zo, 1, &h_emlrtRTEI, true);
  emxInit_real_T(&cpu_c, 1, &i_emlrtRTEI, true);
  emxInit_real_T(&cpu_yc3, 1, &j_emlrtRTEI, true);
  emxInit_real_T(&cpu__2, 1, &h_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_zo, 1, &h_emlrtRTEI, true);
  emxInit_real_T(&cpu_yc5, 1, &k_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_b, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_a, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_z, 1, &e_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_b, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_a, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_b, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_a, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&e_cpu_b, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&e_cpu_a, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_yc3, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_z, 1, &e_emlrtRTEI, true);
  emxInit_real_T(&f_cpu_b, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&f_cpu_a, 1, &d_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_yc2, 1, &d_emlrtRTEI, true);
  for (int32_T ch{0}; ch < b_i; ch++) {
    int32_T i5;
    boolean_T validLaunchParams;
    // 'ec_filtfilt1:103' y(:,ch) = filtfilt_lfn(x(:,ch),b,a,z,nf,L);
    i = cpu_dv->size[0];
    cpu_dv->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real_T(cpu_dv, i, &b_emlrtRTEI);
    i = cpu_x->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (x_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
    if (x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_filtfilt1_kernel1<<<grid, block>>>(ch, gpu_x, i, gpu_dv,
                                            cpu_x->size[0U]);
    }
    dv_outdatedOnGpu = false;
    //  oneCh_lfn %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //  Perform filtering of input data with no phase distortion
    //
    //  xc: matrix of input data
    //  yc: matrix of output data, same dimensions as xc
    //  a,b: IIR coefficients, both of same order/length
    //  zi: initial states
    //  nfact: scalar
    //  L: scalar
    //  odt: output data type
    //
    //  Same comments as in ffOneChan, except here we need to use bsxfun.
    //  Instead of doing scalar subtraction with a vector, we are doing
    //  vector addition with a matrix.  bsxfun replicates the vector
    //  for us.
    //
    //  We also take care to preserve column dimensions
    // coder.gpu.nokernel(); % No kernel due to dependencies across loops
    // coder.gpu.kernelfun; % Add kernelfun pragma to trigger kernel creation
    //  Loop by L
    // 'ec_filtfilt1:131' for ii = 1:L
    for (int32_T ii{0}; ii < i1; ii++) {
      real_T a;
      int32_T bcoef;
      int32_T i6;
      int32_T i7;
      boolean_T dv_outdatedOnCpu;
      // 'ec_filtfilt1:132' xt = bsxfun(@minus, 2*x(1,:),x(nf(1,1)+1:-1:2,:));
      gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
      dv_outdatedOnCpu = false;
      a = 2.0 * cpu_dv->data[0];
      xt_outdatedOnCpu = false;
      yc3_outdatedOnCpu = false;
      i = cpu_xt->size[0];
      cpu_xt->size[0] = csz_idx_0;
      emxEnsureCapacity_real_T(cpu_xt, i, &c_emlrtRTEI);
      if (csz_idx_0 != 0) {
        mwGetLaunchParameters1D(computeNumIters(csz_idx_0 - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
        gpuEmxEnsureCapacity_real_T(cpu_xt, &gpu_xt, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_filtfilt1_kernel2<<<grid, block>>>(
              gpu_dv, static_cast<int32_T>(div_s32(i4 - i2, i3) + 1 != 1), i3,
              i2, a, csz_idx_0 - 1, gpu_xt);
        }
        xt_outdatedOnCpu = true;
      }
      // 'ec_filtfilt1:133' [~,zo] =
      // filter(b(:,ii),a(:,ii),xt,z(:,ii)*xt(1,:),1);
      if (xt_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_xt, &gpu_xt);
      }
      a = cpu_xt->data[0];
      i = b_cpu_b->size[0];
      b_cpu_b->size[0] = cpu_b->size[0];
      emxEnsureCapacity_real_T(b_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (zo_outdatedOnGpu) {
        gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, !b_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_real_T(b_cpu_b, &b_gpu_b, true);
      if (b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_b, cpu_b);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel3<<<grid, block>>>(gpu_b, i, b_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[0];
      emxEnsureCapacity_real_T(b_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (yc3_outdatedOnGpu) {
        gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, !a_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_real_T(b_cpu_a, &b_gpu_a, true);
      if (a_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel4<<<grid, block>>>(gpu_a, i, b_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      i = b_cpu_z->size[0];
      b_cpu_z->size[0] = cpu_z->size[0];
      emxEnsureCapacity_real_T(b_cpu_z, i, &e_emlrtRTEI);
      i = cpu_z->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (z_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_real_T(b_cpu_z, &b_gpu_z, true);
      if (z_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_z, cpu_z);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel5<<<grid, block>>>(a, gpu_z, i, b_gpu_z);
      }
      z_outdatedOnGpu = false;
      yc2_outdatedOnCpu = true;
      coder::filter(b_cpu_b, &b_outdatedOnCpu, &b_gpu_b, &b_outdatedOnGpu,
                    b_cpu_a, &a_outdatedOnCpu, &b_gpu_a, &a_outdatedOnGpu,
                    cpu_xt, &gpu_xt, &yc3_outdatedOnCpu, b_cpu_z,
                    &yc2_outdatedOnCpu, &b_gpu_z, &z_outdatedOnGpu, cpu__1,
                    &yc5_outdatedOnCpu, &gpu__1, &yc5_outdatedOnGpu, cpu_zo,
                    &yc3_outdatedOnGpu, &gpu_zo, &zo_outdatedOnGpu);
      //  outer product
      // 'ec_filtfilt1:134' [yc2,zo] = filter(b(:,ii),a(:,ii),x,zo,1);
      i = c_cpu_b->size[0];
      c_cpu_b->size[0] = cpu_b->size[0];
      emxEnsureCapacity_real_T(c_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(c_cpu_b, &c_gpu_b, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel6<<<grid, block>>>(gpu_b, i, c_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = c_cpu_a->size[0];
      c_cpu_a->size[0] = cpu_a->size[0];
      emxEnsureCapacity_real_T(c_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(c_cpu_a, &c_gpu_a, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel7<<<grid, block>>>(gpu_a, i, c_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      coder::filter(c_cpu_b, &b_outdatedOnCpu, &c_gpu_b, &b_outdatedOnGpu,
                    c_cpu_a, &a_outdatedOnCpu, &c_gpu_a, &a_outdatedOnGpu,
                    cpu_dv, &gpu_dv, &dv_outdatedOnGpu, cpu_zo,
                    &yc3_outdatedOnGpu, &gpu_zo, &zo_outdatedOnGpu, cpu_yc2,
                    &yc2_outdatedOnCpu, &gpu_yc2, &yc2_outdatedOnGpu, b_cpu_zo,
                    &zo_outdatedOnCpu, &b_gpu_zo, &b_zo_outdatedOnGpu);
      // 'ec_filtfilt1:135' xt = bsxfun(@minus,
      // 2*x(end,:),x(end-1:-1:end-nf(1,1),:));
      a = static_cast<real_T>(cpu_dv->size[0]) - nf;
      if (a > static_cast<real_T>(cpu_dv->size[0]) - 1.0) {
        i6 = 1;
        i7 = 1;
        i5 = 0;
      } else {
        i6 = cpu_dv->size[0] - 1;
        i7 = -1;
        i5 = static_cast<int32_T>(a);
      }
      // 'ec_filtfilt1:136' yc3 = filter(b(:,ii),a(:,ii),xt,zo,1);
      if (dv_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
      }
      a = 2.0 * cpu_dv->data[cpu_dv->size[0] - 1];
      i = div_s32(i5 - i6, i7) + 1;
      yc5_outdatedOnCpu = false;
      yc5_outdatedOnGpu = false;
      bcoef = cpu_c->size[0];
      cpu_c->size[0] = i;
      emxEnsureCapacity_real_T(cpu_c, bcoef, &c_emlrtRTEI);
      if (i != 0) {
        mwGetLaunchParameters1D(computeNumIters(i - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, !dv_outdatedOnGpu);
        gpuEmxEnsureCapacity_real_T(cpu_c, &gpu_c, true);
        if (dv_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_dv, cpu_dv);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_filtfilt1_kernel8<<<grid, block>>>(
              gpu_dv, static_cast<int32_T>(div_s32(i5 - i6, i7) + 1 != 1), i7,
              i6, a, i - 1, gpu_c);
        }
        yc5_outdatedOnCpu = true;
      }
      i = d_cpu_b->size[0];
      d_cpu_b->size[0] = cpu_b->size[0];
      emxEnsureCapacity_real_T(d_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(d_cpu_b, &d_gpu_b, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel9<<<grid, block>>>(gpu_b, i, d_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = d_cpu_a->size[0];
      d_cpu_a->size[0] = cpu_a->size[0];
      emxEnsureCapacity_real_T(d_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(d_cpu_a, &d_gpu_a, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel10<<<grid, block>>>(gpu_a, i, d_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      coder::b_filter(d_cpu_b, &b_outdatedOnCpu, &d_gpu_b, &b_outdatedOnGpu,
                      d_cpu_a, &a_outdatedOnCpu, &d_gpu_a, &a_outdatedOnGpu,
                      cpu_c, &gpu_c, &yc5_outdatedOnGpu, b_cpu_zo,
                      &zo_outdatedOnCpu, &b_gpu_zo, &b_zo_outdatedOnGpu,
                      cpu_yc3, &yc3_outdatedOnCpu, &gpu_yc3,
                      &yc3_outdatedOnGpu);
      // 'ec_filtfilt1:138' [~,zo] =
      // filter(b(:,ii),a(:,ii),yc3(end:-1:1,:),z(:,ii)*yc3(end,:),1);
      if (cpu_yc3->size[0] < 1) {
        i5 = 0;
        bcoef = 1;
        i6 = -1;
      } else {
        i5 = cpu_yc3->size[0] - 1;
        bcoef = -1;
        i6 = 0;
      }
      if (yc3_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_yc3, &gpu_yc3);
      }
      a = cpu_yc3->data[cpu_yc3->size[0] - 1];
      i = e_cpu_b->size[0];
      e_cpu_b->size[0] = cpu_b->size[0];
      emxEnsureCapacity_real_T(e_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(e_cpu_b, &e_gpu_b, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel11<<<grid, block>>>(gpu_b, i, e_gpu_b);
      }
      b_outdatedOnGpu = false;
      b_outdatedOnCpu = true;
      i = e_cpu_a->size[0];
      e_cpu_a->size[0] = cpu_a->size[0];
      emxEnsureCapacity_real_T(e_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(e_cpu_a, &e_gpu_a, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel12<<<grid, block>>>(gpu_a, i, e_gpu_a);
      }
      a_outdatedOnGpu = false;
      a_outdatedOnCpu = true;
      i = b_cpu_yc3->size[0];
      b_cpu_yc3->size[0] = div_s32(i6 - i5, bcoef) + 1;
      emxEnsureCapacity_real_T(b_cpu_yc3, i, &d_emlrtRTEI);
      i = (i6 - i5) / bcoef;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_yc3, &gpu_yc3, !yc3_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(b_cpu_yc3, &b_gpu_yc3, true);
      if (yc3_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_yc3, cpu_yc3);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel13<<<grid, block>>>(gpu_yc3, bcoef, i5, i,
                                               b_gpu_yc3);
      }
      yc3_outdatedOnGpu = false;
      i = c_cpu_z->size[0];
      c_cpu_z->size[0] = cpu_z->size[0];
      emxEnsureCapacity_real_T(c_cpu_z, i, &e_emlrtRTEI);
      i = cpu_z->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      z_needsGpuEnsureCapacity = false;
      gpuEmxEnsureCapacity_real_T(c_cpu_z, &c_gpu_z, true);
      z_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel14<<<grid, block>>>(a, gpu_z, i, c_gpu_z);
      }
      xt_outdatedOnCpu = false;
      yc2_outdatedOnCpu = true;
      coder::filter(e_cpu_b, &b_outdatedOnCpu, &e_gpu_b, &b_outdatedOnGpu,
                    e_cpu_a, &a_outdatedOnCpu, &e_gpu_a, &a_outdatedOnGpu,
                    b_cpu_yc3, &b_gpu_yc3, &yc3_outdatedOnGpu, c_cpu_z,
                    &yc2_outdatedOnCpu, &c_gpu_z, &xt_outdatedOnCpu, cpu__2,
                    &yc5_outdatedOnCpu, &gpu__2, &yc5_outdatedOnGpu, c_cpu_zo,
                    &zo_outdatedOnCpu, &c_gpu_zo, &b_zo_outdatedOnGpu);
      //  outer product
      // 'ec_filtfilt1:139' yc5 = filter(b(:,ii),a(:,ii),yc2(end:-1:1,:),zo,1);
      if (cpu_yc2->size[0] < 1) {
        i5 = 0;
        bcoef = 1;
        i6 = -1;
      } else {
        i5 = cpu_yc2->size[0] - 1;
        bcoef = -1;
        i6 = 0;
      }
      i = f_cpu_b->size[0];
      f_cpu_b->size[0] = cpu_b->size[0];
      emxEnsureCapacity_real_T(f_cpu_b, i, &d_emlrtRTEI);
      i = cpu_b->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      zo_outdatedOnGpu = false;
      gpuEmxEnsureCapacity_real_T(f_cpu_b, &f_gpu_b, true);
      b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel15<<<grid, block>>>(gpu_b, i, f_gpu_b);
      }
      xt_outdatedOnCpu = false;
      b_outdatedOnCpu = true;
      i = f_cpu_a->size[0];
      f_cpu_a->size[0] = cpu_a->size[0];
      emxEnsureCapacity_real_T(f_cpu_a, i, &d_emlrtRTEI);
      i = cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      yc3_outdatedOnGpu = false;
      gpuEmxEnsureCapacity_real_T(f_cpu_a, &f_gpu_a, true);
      a_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel16<<<grid, block>>>(gpu_a, i, f_gpu_a);
      }
      yc3_outdatedOnCpu = false;
      a_outdatedOnCpu = true;
      i = b_cpu_yc2->size[0];
      b_cpu_yc2->size[0] = div_s32(i6 - i5, bcoef) + 1;
      emxEnsureCapacity_real_T(b_cpu_yc2, i, &d_emlrtRTEI);
      i = (i6 - i5) / bcoef;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_yc2, &gpu_yc2, !yc2_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(b_cpu_yc2, &b_gpu_yc2, true);
      if (yc2_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_yc2, cpu_yc2);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel17<<<grid, block>>>(gpu_yc2, bcoef, i5, i,
                                               b_gpu_yc2);
      }
      yc2_outdatedOnGpu = false;
      coder::b_filter(f_cpu_b, &b_outdatedOnCpu, &f_gpu_b, &xt_outdatedOnCpu,
                      f_cpu_a, &a_outdatedOnCpu, &f_gpu_a, &yc3_outdatedOnCpu,
                      b_cpu_yc2, &b_gpu_yc2, &yc2_outdatedOnGpu, c_cpu_zo,
                      &zo_outdatedOnCpu, &c_gpu_zo, &b_zo_outdatedOnGpu,
                      cpu_yc5, &yc5_outdatedOnCpu, &gpu_yc5,
                      &yc5_outdatedOnGpu);
      // 'ec_filtfilt1:141' x = yc5(end:-1:1,:);
      if (cpu_yc5->size[0] < 1) {
        i5 = 0;
        bcoef = 1;
        i6 = -1;
      } else {
        i5 = cpu_yc5->size[0] - 1;
        bcoef = -1;
        i6 = 0;
      }
      i = cpu_dv->size[0];
      cpu_dv->size[0] = div_s32(i6 - i5, bcoef) + 1;
      emxEnsureCapacity_real_T(cpu_dv, i, &d_emlrtRTEI);
      i = (i6 - i5) / bcoef;
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_yc5, &gpu_yc5, !yc5_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
      if (yc5_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_yc5, cpu_yc5);
      }
      yc5_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_filtfilt1_kernel18<<<grid, block>>>(gpu_yc5, bcoef, i5, i, gpu_dv);
      }
      dv_outdatedOnGpu = false;
    }
    //  LocalWords:  x b a nf L
    //  LocalWords:  Lx th zi nfilt xc yc IIR nfact xt unreversed nullcopy Npts
    //  nb na yout LocalWords:  Nchans filtord designfilt noisysignals
    //  highpassiir Sanjit Mitra LocalWords:  nd Graw Fredrik Gustafsson
    i5 = cpu_y->size[0];
    mwGetLaunchParameters1D(computeNumIters(i5 - 1), &grid, &block,
                            2147483647U);
    if (y_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, true);
    }
    y_needsGpuEnsureCapacity = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_filtfilt1_kernel19<<<grid, block>>>(gpu_dv, ch, i5 - 1, gpu_y,
                                             cpu_y->size[0U]);
    }
    yc5_outdatedOnCpu = true;
  }
  emxFree_real_T(&b_cpu_yc2);
  emxFree_real_T(&f_cpu_a);
  emxFree_real_T(&f_cpu_b);
  emxFree_real_T(&c_cpu_z);
  emxFree_real_T(&b_cpu_yc3);
  emxFree_real_T(&e_cpu_a);
  emxFree_real_T(&e_cpu_b);
  emxFree_real_T(&d_cpu_a);
  emxFree_real_T(&d_cpu_b);
  emxFree_real_T(&c_cpu_a);
  emxFree_real_T(&c_cpu_b);
  emxFree_real_T(&b_cpu_z);
  emxFree_real_T(&b_cpu_a);
  emxFree_real_T(&b_cpu_b);
  emxFree_real_T(&cpu_yc5);
  emxFree_real_T(&c_cpu_zo);
  emxFree_real_T(&cpu__2);
  emxFree_real_T(&cpu_yc3);
  emxFree_real_T(&cpu_c);
  emxFree_real_T(&b_cpu_zo);
  emxFree_real_T(&cpu_yc2);
  emxFree_real_T(&cpu_zo);
  emxFree_real_T(&cpu__1);
  emxFree_real_T(&cpu_xt);
  emxFree_real_T(&cpu_dv);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (yc5_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &gpu_y);
  }
  gpuEmxFree_real_T(&gpu_x);
  gpuEmxFree_real_T(&gpu_b);
  gpuEmxFree_real_T(&gpu_a);
  gpuEmxFree_real_T(&gpu_z);
  gpuEmxFree_real_T(&gpu_y);
  gpuEmxFree_real_T(&gpu_dv);
  gpuEmxFree_real_T(&gpu_xt);
  gpuEmxFree_real_T(&gpu__1);
  gpuEmxFree_real_T(&gpu_zo);
  gpuEmxFree_real_T(&gpu_yc2);
  gpuEmxFree_real_T(&b_gpu_zo);
  gpuEmxFree_real_T(&gpu_c);
  gpuEmxFree_real_T(&gpu_yc3);
  gpuEmxFree_real_T(&gpu__2);
  gpuEmxFree_real_T(&c_gpu_zo);
  gpuEmxFree_real_T(&gpu_yc5);
  gpuEmxFree_real_T(&b_gpu_b);
  gpuEmxFree_real_T(&b_gpu_a);
  gpuEmxFree_real_T(&b_gpu_z);
  gpuEmxFree_real_T(&c_gpu_b);
  gpuEmxFree_real_T(&c_gpu_a);
  gpuEmxFree_real_T(&d_gpu_b);
  gpuEmxFree_real_T(&d_gpu_a);
  gpuEmxFree_real_T(&e_gpu_b);
  gpuEmxFree_real_T(&e_gpu_a);
  gpuEmxFree_real_T(&b_gpu_yc3);
  gpuEmxFree_real_T(&c_gpu_z);
  gpuEmxFree_real_T(&f_gpu_b);
  gpuEmxFree_real_T(&f_gpu_a);
  gpuEmxFree_real_T(&b_gpu_yc2);
}

// End of code generation (ec_filtfilt1.cu)

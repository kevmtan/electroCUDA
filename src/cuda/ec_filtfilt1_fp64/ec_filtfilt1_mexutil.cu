//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_filtfilt1_mexutil.cu
//
// Code generation for function 'ec_filtfilt1_mexutil'
//

// Include files
#include "ec_filtfilt1_mexutil.h"
#include "ec_filtfilt1_data.h"
#include "ec_filtfilt1_types.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cstring>

// Function Definitions
void checkCudaError(hipError_t errCode, const char_T *file, uint32_T b_line)
{
  if (errCode != hipSuccess) {
    raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                   hipGetErrorString(errCode));
  }
}

uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  return numIters;
}

void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                 emxArray_real_T *gpu, boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                 const emxArray_real_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
}

void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                    const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  len = strlen(file);
  pn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  fn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
}

// End of code generation (ec_filtfilt1_mexutil.cu)

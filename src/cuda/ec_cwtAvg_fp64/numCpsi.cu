#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// numCpsi.cu
//
// Code generation for function 'numCpsi'
//

// Include files
#include "numCpsi.h"
#include "cospiAndSinpi.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "quadgk.h"
#include "rt_nonfinite.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo ad_emlrtRTEI{
    28,        // lineNo
    47,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo bd_emlrtRTEI{
    28,        // lineNo
    36,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo cd_emlrtRTEI{
    28,        // lineNo
    23,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo dd_emlrtRTEI{
    28,        // lineNo
    46,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo ed_emlrtRTEI{
    1,         // lineNo
    17,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    28,        // lineNo
    19,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
real_T numCpsi(const char_T wname[5], real_T varargin_1, real_T varargin_2)
{
  static const real_T gam[23]{1.0,
                              1.0,
                              2.0,
                              6.0,
                              24.0,
                              120.0,
                              720.0,
                              5040.0,
                              40320.0,
                              362880.0,
                              3.6288E+6,
                              3.99168E+7,
                              4.790016E+8,
                              6.2270208E+9,
                              8.71782912E+10,
                              1.307674368E+12,
                              2.0922789888E+13,
                              3.55687428096E+14,
                              6.402373705728E+15,
                              1.21645100408832E+17,
                              2.43290200817664E+18,
                              5.109094217170944E+19,
                              1.1240007277776077E+21};
  static const real_T p[8]{-1.716185138865495,  24.76565080557592,
                           -379.80425647094563, 629.3311553128184,
                           866.96620279041326,  -31451.272968848367,
                           -36144.413418691176, 66456.143820240541};
  static const real_T q[8]{-30.840230011973897, 315.35062697960416,
                           -1015.1563674902192, -3107.7716715723109,
                           22538.11842098015,   4755.8462775278813,
                           -134659.95986496931, -115132.25967555349};
  static const char_T cv1[5]{'m', 'o', 'r', 's', 'e'};
  static const char_T vstr[5]{'m', 'o', 'r', 's', 'e'};
  emxArray_real_T *a;
  emxArray_real_T *b_a;
  emxArray_real_T *b_x;
  emxArray_real_T *c_x;
  emxArray_real_T *d_x;
  emxArray_real_T *e_x;
  emxArray_real_T *f_x;
  emxArray_real_T *fx;
  emxArray_real_T *xt;
  emxArray_real_T *y;
  real_T interval[650];
  real_T absxk;
  real_T anorm;
  real_T cpsi;
  real_T pathlen;
  int32_T exitg1;
  int32_T ix;
  int32_T nx;
  char_T partial_match[5];
  char_T wavname[5];
  boolean_T first_iteration;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nx = 0;
  first_iteration = false;
  ix = 0;
  do {
    exitg1 = 0;
    if (ix + 1 < 6) {
      if (cv[static_cast<uint8_T>(wname[ix]) & 127] !=
          cv[static_cast<int32_T>(cv1[ix])]) {
        exitg1 = 1;
      } else {
        ix++;
      }
    } else {
      first_iteration = true;
      exitg1 = 1;
    }
  } while (exitg1 == 0);
  if (first_iteration) {
    nx = 1;
    for (ix = 0; ix < 5; ix++) {
      partial_match[ix] = vstr[ix];
    }
  } else {
    for (ix = 0; ix < 5; ix++) {
      partial_match[ix] = ' ';
    }
  }
  if (nx != 0) {
    for (ix = 0; ix < 5; ix++) {
      wavname[ix] = partial_match[ix];
    }
  }
  anorm = 2.0 * std::exp(varargin_2 / varargin_1 *
                         ((std::log(varargin_1) - std::log(varargin_2)) + 1.0));
  first_iteration = false;
  ix = 0;
  do {
    exitg1 = 0;
    if (ix + 1 < 6) {
      if (cv[static_cast<int32_T>(wavname[ix])] !=
          cv[static_cast<int32_T>(cv1[ix])]) {
        exitg1 = 1;
      } else {
        ix++;
      }
    } else {
      first_iteration = true;
      exitg1 = 1;
    }
  } while (exitg1 == 0);
  if (first_iteration) {
    real_T x;
    x = 2.0 * varargin_2 / varargin_1;
    if ((x >= 1.0) && (x <= 23.0) && (x == std::floor(x))) {
      x = gam[static_cast<int32_T>(x) - 1];
    } else if ((x < 1.0) && (x == std::floor(x))) {
      x = rtInf;
    } else if (!std::isnan(x)) {
      if (std::isinf(x)) {
        x = rtInf;
      } else {
        real_T midpt;
        real_T tol;
        tol = 1.0;
        nx = 1;
        first_iteration = false;
        if (x <= 0.0) {
          midpt = std::floor(-x);
          first_iteration = (midpt != std::floor(-x / 2.0) * 2.0);
          ::coder::internal::scalar::cospiAndSinpi(-x - midpt, &absxk);
          tol = -3.1415926535897931 / absxk;
          x = -x + 1.0;
        }
        if (x < 12.0) {
          real_T abserrsubk;
          real_T halfh;
          abserrsubk = x;
          if (x < 1.0) {
            midpt = x;
            x++;
          } else {
            nx = static_cast<int32_T>(std::floor(x));
            x -= std::floor(x) - 1.0;
            midpt = x - 1.0;
          }
          absxk = 0.0 * midpt;
          halfh = 1.0;
          for (ix = 0; ix < 8; ix++) {
            absxk = (absxk + p[ix]) * midpt;
            halfh = halfh * midpt + q[ix];
          }
          midpt = absxk / halfh + 1.0;
          if (abserrsubk < x) {
            midpt /= abserrsubk;
          } else if (abserrsubk > x) {
            for (int32_T j{0}; j <= nx - 2; j++) {
              midpt *= x;
              x++;
            }
          }
        } else {
          real_T halfh;
          midpt = x * x;
          halfh = 0.0057083835261;
          for (ix = 0; ix < 6; ix++) {
            halfh = halfh / midpt + dv3[ix];
          }
          halfh = (halfh / x - x) + 0.91893853320467278;
          halfh += (x - 0.5) * std::log(x);
          midpt = std::exp(halfh);
        }
        if (first_iteration) {
          midpt = -midpt;
        }
        if (tol != 1.0) {
          midpt = tol / midpt;
        }
        x = midpt;
      }
    }
    cpsi = anorm * anorm / (2.0 * varargin_1) *
           rt_powd_snf(0.5, 2.0 * (varargin_2 / varargin_1) - 1.0) * x;
  } else {
    real_T subs[1298];
    real_T errsub[649];
    real_T qsub[649];
    interval[0] = -1.0;
    interval[1] = 1.0;
    std::memset(&interval[2], 0, 648U * sizeof(real_T));
    cpsi = 0.0;
    ix = split(interval, 2, &pathlen);
    if (!(pathlen > 0.0)) {
      cpsi = 0.47999999999999976;
    } else {
      real_T err_ok;
      real_T q_ok;
      int32_T k;
      int32_T nsubs;
      nsubs = ix - 2;
      for (k = 0; k <= nsubs; k++) {
        subs[k << 1] = interval[k];
        subs[(k << 1) + 1] = interval[k + 1];
      }
      q_ok = 0.0;
      err_ok = 0.0;
      first_iteration = true;
      emxInit_real_T(&b_x, 2, &hc_emlrtRTEI, true);
      emxInit_real_T(&c_x, 2, &nc_emlrtRTEI, true);
      emxInit_real_T(&xt, 2, &oc_emlrtRTEI, true);
      emxInit_real_T(&y, 2, &dd_emlrtRTEI, true);
      emxInit_real_T(&a, 2, &ad_emlrtRTEI, true);
      emxInit_real_T(&fx, 2, &ed_emlrtRTEI, true);
      emxInit_real_T(&d_x, 2, &bd_emlrtRTEI, true);
      emxInit_real_T(&b_a, 2, &ad_emlrtRTEI, true);
      emxInit_real_T(&e_x, 2, &cd_emlrtRTEI, true);
      emxInit_real_T(&f_x, 2, &fd_emlrtRTEI, true);
      do {
        real_T halfh;
        real_T midpt;
        boolean_T guard1;
        exitg1 = 0;
        ix = b_x->size[0] * b_x->size[1];
        b_x->size[0] = 1;
        b_x->size[1] = 15 * (nsubs + 1);
        emxEnsureCapacity_real_T(b_x, ix, &hc_emlrtRTEI);
        ix = -1;
        for (k = 0; k <= nsubs; k++) {
          anorm = subs[k << 1];
          absxk = subs[(k << 1) + 1];
          midpt = (anorm + absxk) / 2.0;
          halfh = (absxk - anorm) / 2.0;
          for (int32_T j{0}; j < 15; j++) {
            b_x->data[(ix + j) + 1] = dv[j] * halfh + midpt;
          }
          ix += 15;
        }
        ix = c_x->size[0] * c_x->size[1];
        c_x->size[0] = 1;
        c_x->size[1] = b_x->size[1];
        emxEnsureCapacity_real_T(c_x, ix, &ic_emlrtRTEI);
        ix = xt->size[0] * xt->size[1];
        xt->size[0] = 1;
        xt->size[1] = b_x->size[1];
        emxEnsureCapacity_real_T(xt, ix, &jc_emlrtRTEI);
        ix = b_x->size[1];
        for (k = 0; k < ix; k++) {
          anorm = b_x->data[k];
          midpt = anorm * anorm;
          c_x->data[k] = 0.29999999999999982 * anorm * (3.0 - midpt) + 5.0;
          xt->data[k] = 0.89999999999999947 * (1.0 - midpt);
        }
        guard1 = false;
        if (!first_iteration) {
          boolean_T exitg2;
          absxk = std::abs(c_x->data[0]);
          k = 0;
          exitg2 = false;
          while ((!exitg2) && (k <= c_x->size[1] - 2)) {
            real_T x;
            x = absxk;
            absxk = std::abs(c_x->data[k + 1]);
            if (std::abs(c_x->data[k + 1] - c_x->data[k]) <=
                2.2204460492503131E-14 * std::fmax(x, absxk)) {
              first_iteration = true;
              exitg2 = true;
            } else {
              k++;
            }
          }
          if (first_iteration) {
            int16_T dv_idx_1;
            dv_idx_1 = static_cast<int16_T>(b_x->size[1]);
            ix = fx->size[0] * fx->size[1];
            fx->size[0] = 1;
            fx->size[1] = b_x->size[1];
            emxEnsureCapacity_real_T(fx, ix, &lc_emlrtRTEI);
            for (ix = 0; ix < dv_idx_1; ix++) {
              fx->data[ix] = 0.0;
            }
          } else {
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1) {
          first_iteration = false;
          ix = a->size[0] * a->size[1];
          a->size[0] = 1;
          a->size[1] = c_x->size[1];
          emxEnsureCapacity_real_T(a, ix, &ad_emlrtRTEI);
          for (ix = 0; ix < c_x->size[1]; ix++) {
            a->data[ix] = (c_x->data[ix] - 5.0) / 0.6;
          }
          ix = y->size[0] * y->size[1];
          y->size[0] = 1;
          y->size[1] = a->size[1];
          emxEnsureCapacity_real_T(y, ix, &m_emlrtRTEI);
          nx = a->size[1];
          for (k = 0; k < nx; k++) {
            anorm = a->data[k];
            y->data[k] = anorm * anorm;
          }
          ix = d_x->size[0] * d_x->size[1];
          d_x->size[0] = 1;
          d_x->size[1] = y->size[1];
          emxEnsureCapacity_real_T(d_x, ix, &bd_emlrtRTEI);
          for (ix = 0; ix < y->size[1]; ix++) {
            d_x->data[ix] = -2.0 * (1.0 / (1.0 - y->data[ix]));
          }
          ix = b_a->size[0] * b_a->size[1];
          b_a->size[0] = 1;
          b_a->size[1] = c_x->size[1];
          emxEnsureCapacity_real_T(b_a, ix, &ad_emlrtRTEI);
          for (ix = 0; ix < c_x->size[1]; ix++) {
            b_a->data[ix] = (c_x->data[ix] - 5.0) / 0.6;
          }
          nx = b_a->size[1];
          for (k = 0; k < nx; k++) {
            d_x->data[k] = std::exp(d_x->data[k]);
          }
          ix = e_x->size[0] * e_x->size[1];
          e_x->size[0] = 1;
          e_x->size[1] = d_x->size[1];
          emxEnsureCapacity_real_T(e_x, ix, &cd_emlrtRTEI);
          for (ix = 0; ix < d_x->size[1]; ix++) {
            e_x->data[ix] = 14.7781121978613 * d_x->data[ix];
          }
          nx = d_x->size[1];
          ix = f_x->size[0] * f_x->size[1];
          f_x->size[0] = 1;
          f_x->size[1] = e_x->size[1];
          emxEnsureCapacity_real_T(f_x, ix, &l_emlrtRTEI);
          for (k = 0; k < nx; k++) {
            f_x->data[k] = std::abs(e_x->data[k]);
          }
          if (f_x->size[1] == c_x->size[1]) {
            ix = fx->size[0] * fx->size[1];
            fx->size[0] = 1;
            fx->size[1] = f_x->size[1];
            emxEnsureCapacity_real_T(fx, ix, &mc_emlrtRTEI);
            for (ix = 0; ix < f_x->size[1]; ix++) {
              fx->data[ix] = f_x->data[ix] / c_x->data[ix] * xt->data[ix];
            }
          } else {
            binary_expand_op_7(fx, f_x, c_x, xt);
          }
        }
        if (first_iteration) {
          exitg1 = 1;
        } else {
          real_T tol;
          midpt = 0.0;
          ix = -1;
          for (k = 0; k <= nsubs; k++) {
            anorm = 0.0;
            absxk = 0.0;
            for (int32_T j{0}; j < 15; j++) {
              anorm += dv1[j] * fx->data[(ix + j) + 1];
              absxk += dv2[j] * fx->data[(ix + j) + 1];
            }
            ix += 15;
            halfh = (subs[(k << 1) + 1] - subs[k << 1]) / 2.0;
            anorm *= halfh;
            qsub[k] = anorm;
            midpt += anorm;
            errsub[k] = absxk * halfh;
          }
          cpsi = midpt + q_ok;
          tol = std::fmax(1.0E-10, 1.0E-6 * std::abs(cpsi));
          absxk = 2.0 * tol / pathlen;
          midpt = 0.0;
          ix = 0;
          for (k = 0; k <= nsubs; k++) {
            real_T abserrsubk;
            anorm = errsub[k];
            abserrsubk = std::abs(anorm);
            if (abserrsubk <=
                absxk * ((subs[(k << 1) + 1] - subs[k << 1]) / 2.0)) {
              err_ok += anorm;
              q_ok += qsub[k];
            } else {
              midpt += abserrsubk;
              ix++;
              subs[(ix - 1) << 1] = subs[k << 1];
              subs[((ix - 1) << 1) + 1] = subs[(k << 1) + 1];
            }
          }
          halfh = std::abs(err_ok) + midpt;
          if ((!std::isinf(cpsi)) && (!std::isnan(cpsi)) &&
              ((!std::isinf(halfh)) && (!std::isnan(halfh))) && (ix != 0) &&
              (!(halfh <= tol))) {
            nsubs = (ix << 1) - 1;
            if (nsubs + 1 > 650) {
              exitg1 = 1;
            } else {
              for (k = 0; k < ix; k++) {
                subs[((((ix - k) << 1) - 1) << 1) + 1] =
                    subs[(((ix - k) - 1) << 1) + 1];
                subs[(((ix - k) << 1) - 1) << 1] =
                    (subs[((ix - k) - 1) << 1] +
                     subs[(((ix - k) - 1) << 1) + 1]) /
                    2.0;
                subs[((((ix - k) << 1) - 2) << 1) + 1] =
                    subs[(((ix - k) << 1) - 1) << 1];
                subs[(((ix - k) << 1) - 2) << 1] = subs[((ix - k) - 1) << 1];
              }
            }
          } else {
            exitg1 = 1;
          }
        }
      } while (exitg1 == 0);
      emxFree_real_T(&f_x);
      emxFree_real_T(&e_x);
      emxFree_real_T(&b_a);
      emxFree_real_T(&d_x);
      emxFree_real_T(&fx);
      emxFree_real_T(&a);
      emxFree_real_T(&y);
      emxFree_real_T(&xt);
      emxFree_real_T(&c_x);
      emxFree_real_T(&b_x);
    }
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  return cpsi;
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (numCpsi.cu)

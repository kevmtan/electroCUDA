#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// conv2.cu
//
// Code generation for function 'conv2'
//

// Include files
#include "conv2.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo kf_emlrtRTEI{
    75,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo
    lf_emlrtRTEI{
        158,      // lineNo
        24,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo mf_emlrtRTEI{
    1,                  // lineNo
    1,                  // colNo
    "gpu_conv2_kernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+images/gpu_conv2_kernel.p" // pName
};

// Function Declarations
static __global__ void conv2NonSeparable_kernel35(const int32_T b,
                                                  emxArray_real_T c);

static __global__ void conv2NonSeparable_kernel36(const emxArray_real_T c,
                                                  const int32_T b,
                                                  emxArray_real_T b_c);

static __global__ void conv2NonSeparable_kernel37(const int32_T OH,
                                                  emxArray_real_T expanded);

static __global__ void conv2NonSeparable_kernel38(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel39(const emxArray_real_T a,
                                                  const emxArray_int32_T iv,
                                                  const int32_T b,
                                                  emxArray_real_T expanded);

static __global__ void
conv2NonSeparable_kernel40(const emxArray_real_T expanded,
                           const emxArray_int32_T rows, const emxArray_real_T b,
                           const int32_T c, emxArray_real_T b_c,
                           int32_T b_dim0);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel35(
    const int32_T b, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    c.data[i2] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel36(
    const emxArray_real_T c, const int32_T b, emxArray_real_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    b_c.data[i2] = c.data[i2];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel37(
    const int32_T OH, emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    expanded.data[i2] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel38(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    int32_T q1;
    i2 = static_cast<int32_T>(idx);
    q1 = y.data[i2];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[i2] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel39(
    const emxArray_real_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    expanded.data[iv.data[i2]] = a.data[i2];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel40(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_real_T b, const int32_T c, emxArray_real_T b_c,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(c) + 1UL));
    b_cv = 0.0;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[q1 - 1] * b.data[(b_dim0 - m) - 1];
    }
    b_c.data[orow] = b_cv;
  }
}

//
//
namespace coder {
void conv2NonSeparable(const emxArray_real_T *cpu_a, emxArray_real_T *cpu_b,
                       emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
                       emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
                       emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T b_gpu_c;
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *cpu_expanded;
  int32_T k;
  int32_T mc;
  boolean_T validLaunchParams;
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_expanded);
  gpuEmxReset_real_T(&b_gpu_c);
  gpuEmxReset_real_T(&gpu_a);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  k = cpu_c->size[0] * cpu_c->size[1];
  cpu_c->size[0] = mc;
  cpu_c->size[1] = 1;
  emxEnsureCapacity_real_T(cpu_c, k, &kf_emlrtRTEI);
  mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    conv2NonSeparable_kernel35<<<grid, block>>>(mc - 1, *gpu_c);
  }
  emxInit_real_T(&b_cpu_c, 1, &mf_emlrtRTEI, true);
  emxInit_real_T(&cpu_expanded, 2, &kd_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &sd_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &kd_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &kd_emlrtRTEI, true);
  if ((cpu_a->size[0] != 0) && (cpu_b->size[0] != 0)) {
    if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
      real_T alpha1;
      real_T beta1;
      k = b_cpu_c->size[0];
      b_cpu_c->size[0] = 1;
      emxEnsureCapacity_real_T(b_cpu_c, k, &lf_emlrtRTEI);
      alpha1 = 1.0;
      beta1 = 0.0;
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                              1, 1, 1, (double *)&alpha1,
                              (double *)&gpu_a.data[0], 1,
                              (double *)&gpu_b->data[0], 1, (double *)&beta1,
                              (double *)&b_gpu_c.data[0], 1),
                  __FILE__, __LINE__);
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = 1;
      emxEnsureCapacity_real_T(cpu_c, k, &kf_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel36<<<grid, block>>>(b_gpu_c, mc - 1, *gpu_c);
      }
    } else {
      real_T blockDims[3];
      int32_T b_OH;
      int32_T n;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims[3];
      OH = (static_cast<uint32_T>(cpu_a->size[0]) +
            static_cast<uint32_T>(cpu_b->size[0])) -
           1U;
      mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      blockDims[0] = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
          static_cast<real_T>(mc));
      threadDims[0] = static_cast<int8_T>(mc);
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
      if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_b->size[0];
      }
      k = cpu_expanded->size[0] * cpu_expanded->size[1];
      cpu_expanded->size[0] = mc - 1;
      cpu_expanded->size[1] = 1;
      emxEnsureCapacity_real_T(cpu_expanded, k, &kd_emlrtRTEI);
      mc = (static_cast<int32_T>(OH) + cpu_b->size[0]) - 2;
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_expanded, &gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel37<<<grid, block>>>(mc, gpu_expanded);
      }
      n = cpu_a->size[0];
      k = cpu_y->size[0] * cpu_y->size[1];
      cpu_y->size[0] = 1;
      cpu_y->size[1] = cpu_a->size[0];
      emxEnsureCapacity_int32_T(cpu_y, k, &ld_emlrtRTEI);
      cpu_y->data[0] = 1;
      mc = 1;
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_y->data[k + 1] = mc;
      }
      k = cpu_iv->size[0];
      cpu_iv->size[0] = cpu_y->size[1];
      emxEnsureCapacity_int32_T(cpu_iv, k, &kd_emlrtRTEI);
      mc = cpu_y->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
      gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel38<<<grid, block>>>(offsetH, gpu_y, mc, gpu_iv);
      }
      mc = cpu_iv->size[0];
      mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel39<<<grid, block>>>(gpu_a, gpu_iv, mc - 1,
                                                    gpu_expanded);
      }
      n = cpu_b->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_b->size[0];
      emxEnsureCapacity_int32_T(cpu_rows, k, &ld_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      cpu_c->size[1] = 1;
      emxEnsureCapacity_real_T(cpu_c, k, &kd_emlrtRTEI);
      if (blockDims[0] < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims[0]);
      } else {
        OH = MAX_uint32_T;
      }
      mwApplyLaunchParameters(
          computeNumIters(0, b_OH - 1), dim3(OH, 1U, 1U),
          dim3(static_cast<uint32_T>(threadDims[0]), 1U, 1U), &grid, &block);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel40<<<grid, block>>>(
            gpu_expanded, gpu_rows, *gpu_b, b_OH - 1, *gpu_c, cpu_b->size[0U]);
      }
    }
  }
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&cpu_y);
  emxFree_real_T(&cpu_expanded);
  emxFree_real_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_a);
  gpuEmxFree_real_T(&b_gpu_c);
  gpuEmxFree_real_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_rows);
  *c_outdatedOnCpu = true;
  *c_outdatedOnGpu = false;
}

} // namespace coder

// End of code generation (conv2.cu)

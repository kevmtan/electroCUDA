#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// scNormalize.cu
//
// Code generation for function 'scNormalize'
//

// Include files
#include "scNormalize.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo of_emlrtRTEI{
    12,                                                               // lineNo
    5,                                                                // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo pf_emlrtRTEI{
    12,            // lineNo
    12,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo qf_emlrtRTEI{
    22,            // lineNo
    32,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo rf_emlrtRTEI{
    12,            // lineNo
    1,             // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo sf_emlrtRTEI{
    26,            // lineNo
    43,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo tf_emlrtRTEI{
    26,            // lineNo
    37,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
void scNormalize(emxArray_creal_T *cfs, real_T cpsi,
                 const emxArray_real_T *scales, real_T normfac)
{
  emxArray_real_T b_gpu_c;
  emxArray_real_T gpu_abscfssq;
  emxArray_real_T gpu_c;
  emxArray_real_T gpu_z;
  emxArray_real_T *a;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *cpu_abscfssq;
  emxArray_real_T *cpu_c;
  emxArray_real_T *cpu_z;
  emxArray_real_T *y;
  emxArray_uint32_T *b_y;
  real_T alpha1;
  real_T b_cpu_z;
  real_T c1;
  real_T *b_gpu_z;
  int32_T acoef;
  int32_T nx;
  int32_T u1;
  boolean_T abscfssq_outdatedOnGpu;
  boolean_T z_needsGpuEnsureCapacity;
  boolean_T z_outdatedOnGpu;
  gpuEmxReset_real_T(&b_gpu_c);
  checkCudaError(mwCudaMalloc(&b_gpu_z, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real_T(&gpu_c);
  gpuEmxReset_real_T(&gpu_z);
  gpuEmxReset_real_T(&gpu_abscfssq);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nx = cfs->size[0] * cfs->size[1];
  emxInit_real_T(&y, 2, &pf_emlrtRTEI, true);
  acoef = y->size[0] * y->size[1];
  y->size[0] = cfs->size[0];
  y->size[1] = cfs->size[1];
  emxEnsureCapacity_real_T(y, acoef, &l_emlrtRTEI);
  for (int32_T k{0}; k < nx; k++) {
    y->data[k] = rt_hypotd_snf(cfs->data[k].re, cfs->data[k].im);
  }
  emxInit_real_T(&a, 2, &qf_emlrtRTEI, true);
  acoef = a->size[0] * a->size[1];
  a->size[0] = y->size[0];
  a->size[1] = y->size[1];
  emxEnsureCapacity_real_T(a, acoef, &m_emlrtRTEI);
  nx = y->size[0] * y->size[1];
  for (int32_T k{0}; k < nx; k++) {
    a->data[k] = y->data[k] * y->data[k];
  }
  emxFree_real_T(&y);
  emxInit_real_T(&cpu_abscfssq, 2, &rf_emlrtRTEI, true);
  abscfssq_outdatedOnGpu = false;
  acoef = cpu_abscfssq->size[0] * cpu_abscfssq->size[1];
  nx = scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (scales->size[1] == 1) {
    cpu_abscfssq->size[0] = a->size[0];
  } else if (a->size[0] == 1) {
    cpu_abscfssq->size[0] = scales->size[1];
  } else if (a->size[0] == scales->size[1]) {
    cpu_abscfssq->size[0] = a->size[0];
  } else {
    cpu_abscfssq->size[0] = u1;
  }
  cpu_abscfssq->size[1] = a->size[1];
  emxEnsureCapacity_real_T(cpu_abscfssq, acoef, &h_emlrtRTEI);
  nx = scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (scales->size[1] == 1) {
    u1 = a->size[0];
  } else if (a->size[0] == 1) {
    u1 = scales->size[1];
  } else if (a->size[0] == scales->size[1]) {
    u1 = a->size[0];
  }
  if ((u1 != 0) && (a->size[1] != 0)) {
    int32_T bcoef;
    nx = (a->size[1] != 1);
    u1 = a->size[1] - 1;
    acoef = (a->size[0] != 1);
    bcoef = (scales->size[1] != 1);
    for (int32_T k{0}; k <= u1; k++) {
      int32_T i1;
      int32_T varargin_2;
      varargin_2 = nx * k;
      i1 = cpu_abscfssq->size[0] - 1;
      for (int32_T b_k{0}; b_k <= i1; b_k++) {
        cpu_abscfssq->data[b_k + cpu_abscfssq->size[0] * k] =
            a->data[acoef * b_k + a->size[0] * varargin_2] /
            scales->data[bcoef * b_k];
        abscfssq_outdatedOnGpu = true;
      }
    }
  }
  emxFree_real_T(&a);
  emxInit_uint32_T(&b_y, 2, &sf_emlrtRTEI, true);
  if (cpu_abscfssq->size[1] < 1) {
    b_y->size[0] = 1;
    b_y->size[1] = 0;
  } else {
    uint32_T u;
    u = static_cast<uint32_T>(cpu_abscfssq->size[1]);
    acoef = b_y->size[0] * b_y->size[1];
    b_y->size[0] = 1;
    b_y->size[1] = cpu_abscfssq->size[1];
    emxEnsureCapacity_uint32_T(b_y, acoef, &eb_emlrtRTEI);
    for (acoef = 0; acoef < static_cast<int32_T>(u); acoef++) {
      b_y->data[acoef] = static_cast<uint32_T>(acoef) + 1U;
    }
  }
  emxInit_real_T(&cpu_z, 1, &tf_emlrtRTEI, true);
  z_outdatedOnGpu = false;
  acoef = cpu_z->size[0];
  cpu_z->size[0] = cpu_abscfssq->size[0];
  emxEnsureCapacity_real_T(cpu_z, acoef, &of_emlrtRTEI);
  z_needsGpuEnsureCapacity = true;
  for (acoef = 0; acoef < cpu_abscfssq->size[0]; acoef++) {
    cpu_z->data[acoef] = 0.0;
    z_outdatedOnGpu = true;
  }
  if (cpu_abscfssq->size[1] > 1) {
    emxInit_real_T(&cpu_c, 1, &r_emlrtRTEI, true);
    if (b_y->size[1] == 1) {
      nx = cpu_abscfssq->size[1];
      acoef = cpu_c->size[0];
      cpu_c->size[0] = cpu_abscfssq->size[1];
      emxEnsureCapacity_real_T(cpu_c, acoef, &r_emlrtRTEI);
      for (acoef = 0; acoef < nx; acoef++) {
        cpu_c->data[acoef] = b_y->data[0];
      }
      c1 = 0.5 * static_cast<real_T>(b_y->data[0]);
      cpu_c->data[0] = c1;
      cpu_c->data[cpu_abscfssq->size[1] - 1] = c1;
    } else {
      acoef = cpu_c->size[0];
      cpu_c->size[0] = cpu_abscfssq->size[1];
      emxEnsureCapacity_real_T(cpu_c, acoef, &q_emlrtRTEI);
      cpu_c->data[0] =
          0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[1]) -
                                    static_cast<int32_T>(b_y->data[0]));
      nx = cpu_abscfssq->size[1];
      for (int32_T k{0}; k <= nx - 3; k++) {
        cpu_c->data[k + 1] =
            0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[k + 2]) -
                                      static_cast<int32_T>(b_y->data[k]));
      }
      cpu_c->data[cpu_abscfssq->size[1] - 1] =
          0.5 * static_cast<real_T>(
                    static_cast<int32_T>(b_y->data[cpu_abscfssq->size[1] - 1]) -
                    static_cast<int32_T>(b_y->data[cpu_abscfssq->size[1] - 2]));
    }
    if (cpu_abscfssq->size[0] >= 1) {
      c1 = 0.0;
      alpha1 = 1.0;
      gpuEmxEnsureCapacity_real_T(cpu_abscfssq, &gpu_abscfssq,
                                  !abscfssq_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(cpu_c, &gpu_c, false);
      gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
      z_needsGpuEnsureCapacity = false;
      if (abscfssq_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_abscfssq, cpu_abscfssq);
      }
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_c, cpu_c);
      if (z_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_z, cpu_z);
      }
      cublasCheck(hipblasDgemv(getCublasGlobalHandle(), HIPBLAS_OP_N,
                              cpu_abscfssq->size[0], cpu_abscfssq->size[1],
                              (double *)&alpha1,
                              (double *)&gpu_abscfssq.data[0],
                              cpu_abscfssq->size[0], (double *)&gpu_c.data[0],
                              1, (double *)&c1, (double *)&gpu_z.data[0], 1),
                  __FILE__, __LINE__);
      z_outdatedOnGpu = false;
    }
    emxFree_real_T(&cpu_c);
  }
  emxFree_uint32_T(&b_y);
  b_cpu_z = 0.0;
  abscfssq_outdatedOnGpu = false;
  if (cpu_z->size[0] <= 1) {
    if ((scales->size[1] == 1) &&
        (std::isinf(scales->data[0]) || std::isnan(scales->data[0]))) {
      b_cpu_z = rtNaN;
    }
  } else {
    emxInit_real_T(&b_cpu_c, 1, &r_emlrtRTEI, true);
    if (scales->size[1] == 1) {
      nx = cpu_z->size[0];
      acoef = b_cpu_c->size[0];
      b_cpu_c->size[0] = cpu_z->size[0];
      emxEnsureCapacity_real_T(b_cpu_c, acoef, &r_emlrtRTEI);
      for (acoef = 0; acoef < nx; acoef++) {
        b_cpu_c->data[acoef] = scales->data[0];
      }
      c1 = 0.5 * scales->data[0];
      b_cpu_c->data[0] = c1;
      b_cpu_c->data[cpu_z->size[0] - 1] = c1;
    } else {
      acoef = b_cpu_c->size[0];
      b_cpu_c->size[0] = cpu_z->size[0];
      emxEnsureCapacity_real_T(b_cpu_c, acoef, &q_emlrtRTEI);
      b_cpu_c->data[0] = 0.5 * (scales->data[1] - scales->data[0]);
      nx = cpu_z->size[0];
      for (int32_T k{0}; k <= nx - 3; k++) {
        b_cpu_c->data[k + 1] = 0.5 * (scales->data[k + 2] - scales->data[k]);
      }
      b_cpu_c->data[cpu_z->size[0] - 1] =
          0.5 *
          (scales->data[cpu_z->size[0] - 1] - scales->data[cpu_z->size[0] - 2]);
    }
    c1 = 0.0;
    alpha1 = 1.0;
    if (z_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, false);
    if (z_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_z, cpu_z);
    }
    gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_c, b_cpu_c);
    checkCudaError(hipMemcpy(b_gpu_z, &b_cpu_z, 8UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
    cublasCheck(hipblasDgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                            cpu_z->size[0], 1, (double *)&alpha1,
                            (double *)&gpu_z.data[0], cpu_z->size[0],
                            (double *)&b_gpu_c.data[0], 1, (double *)&c1,
                            (double *)b_gpu_z, 1),
                __FILE__, __LINE__);
    abscfssq_outdatedOnGpu = true;
    emxFree_real_T(&b_cpu_c);
  }
  emxFree_real_T(&cpu_z);
  if (!(normfac == 0.0)) {
    if (abscfssq_outdatedOnGpu) {
      checkCudaError(hipMemcpy(&b_cpu_z, b_gpu_z, 8UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    c1 = std::sqrt(
        normfac /
        (2.0 / cpsi / static_cast<real_T>(cpu_abscfssq->size[1]) * b_cpu_z));
    for (acoef = 0; acoef < cfs->size[1]; acoef++) {
      for (nx = 0; nx < cfs->size[0]; nx++) {
        cfs->data[nx + cfs->size[0] * acoef].re *= c1;
        cfs->data[nx + cfs->size[0] * acoef].im *= c1;
      }
    }
  }
  emxFree_real_T(&cpu_abscfssq);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_abscfssq);
  gpuEmxFree_real_T(&gpu_z);
  gpuEmxFree_real_T(&gpu_c);
  checkCudaError(mwCudaFree(b_gpu_z), __FILE__, __LINE__);
  gpuEmxFree_real_T(&b_gpu_c);
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (scNormalize.cu)

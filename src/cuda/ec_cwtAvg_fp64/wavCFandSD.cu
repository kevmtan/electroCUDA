#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wavCFandSD.cu
//
// Code generation for function 'wavCFandSD'
//

// Include files
#include "wavCFandSD.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "gammaln.h"
#include "quadgk.h"
#include "rt_nonfinite.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo kc_emlrtRTEI{
    64,                // lineNo
    28,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo pc_emlrtRTEI{
    64,                // lineNo
    31,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo qc_emlrtRTEI{
    1,            // lineNo
    39,           // colNo
    "wavCFandSD", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavCFandSD.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    64,                // lineNo
    12,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
real_T wavCFandSD(char_T wname[5], real_T varargin_1, real_T varargin_2,
                  real_T *sigmaT, real_T *cf)
{
  emxArray_real_T *b_dv1;
  emxArray_real_T *b_x;
  emxArray_real_T *b_y;
  emxArray_real_T *fx;
  emxArray_real_T *x;
  emxArray_real_T *xt;
  emxArray_real_T *y;
  real_T interval[650];
  real_T FourierFactor;
  real_T b_be;
  real_T d;
  real_T d1;
  real_T d2;
  real_T d3;
  real_T d4;
  real_T err_ok;
  int32_T k;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  for (k = 0; k < 5; k++) {
    wname[k] = cv[static_cast<uint8_T>(wname[k]) & 127];
  }
  if (wname[0] == 'm') {
    real_T abserrsubk;
    real_T be;
    real_T halfh;
    real_T intFsq;
    real_T midpt;
    real_T q_ok;
    real_T tol;
    *cf = std::exp(1.0 / varargin_1 *
                   (std::log(varargin_2) - std::log(varargin_1)));
    midpt = 2.0 * (varargin_2 - 1.0);
    halfh = 2.0 * varargin_2;
    abserrsubk = (varargin_2 - 1.0) + varargin_1;
    tol = 2.0 * ((varargin_2 - 1.0) + varargin_1);
    intFsq = 2.0 * varargin_2;
    be = (varargin_2 - 1.0) + varargin_1 / 2.0;
    b_be = 2.0 * ((varargin_2 - 1.0) + varargin_1 / 2.0);
    q_ok = 2.0 * varargin_2;
    d = (2.0 * (varargin_2 - 1.0) + 1.0) / varargin_1;
    gammaln(&d);
    d1 = (2.0 * varargin_2 + 1.0) / varargin_1;
    gammaln(&d1);
    err_ok = (2.0 * ((varargin_2 - 1.0) + varargin_1) + 1.0) / varargin_1;
    gammaln(&err_ok);
    d2 = (2.0 * varargin_2 + 1.0) / varargin_1;
    gammaln(&d2);
    d3 = (2.0 * ((varargin_2 - 1.0) + varargin_1 / 2.0) + 1.0) / varargin_1;
    gammaln(&d3);
    d4 = (2.0 * varargin_2 + 1.0) / varargin_1;
    gammaln(&d4);
    *sigmaT = std::sqrt(
        (std::exp(((((((2.0 * (varargin_2 / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(varargin_2))) -
                        2.0 * ((varargin_2 - 1.0) / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(varargin_2 - 1.0)))) +
                       midpt / varargin_1 *
                           ((std::log(varargin_1) + 1.0) - std::log(midpt))) -
                      halfh / varargin_1 *
                          ((std::log(varargin_1) + 1.0) - std::log(halfh))) +
                     2.0 / varargin_1 * std::log(varargin_2 / varargin_1)) +
                    2.0 * std::log(varargin_2)) +
                   d) -
                  d1) +
         std::exp(((((((2.0 * (varargin_2 / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(varargin_2))) -
                        2.0 * (abserrsubk / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(abserrsubk)))) +
                       tol / varargin_1 *
                           ((std::log(varargin_1) + 1.0) - std::log(tol))) -
                      intFsq / varargin_1 *
                          ((std::log(varargin_1) + 1.0) - std::log(intFsq))) +
                     2.0 / varargin_1 * std::log(varargin_2 / varargin_1)) +
                    2.0 * std::log(varargin_1)) +
                   err_ok) -
                  d2)) -
        std::exp(
            ((((((((2.0 * (varargin_2 / varargin_1 *
                           ((std::log(varargin_1) + 1.0) -
                            std::log(varargin_2))) -
                    2.0 * (be / varargin_1 *
                           ((std::log(varargin_1) + 1.0) - std::log(be)))) +
                   b_be / varargin_1 *
                       ((std::log(varargin_1) + 1.0) - std::log(b_be))) -
                  q_ok / varargin_1 *
                      ((std::log(varargin_1) + 1.0) - std::log(q_ok))) +
                 2.0 / varargin_1 * std::log(varargin_2 / varargin_1)) +
                0.69314718055994529) +
               std::log(varargin_2)) +
              std::log(varargin_1)) +
             d3) -
            d4));
    if (std::isinf(*sigmaT) || std::isnan(*sigmaT)) {
      real_T subs[1298];
      real_T errsub[649];
      real_T qsub[649];
      int32_T ix;
      be = std::exp(1.0 / varargin_1 *
                    (std::log(varargin_2) - std::log(varargin_1)));
      interval[0] = 0.0;
      interval[1] = 1.0;
      std::memset(&interval[2], 0, 648U * sizeof(real_T));
      intFsq = 0.0;
      ix = split(interval, 2, &b_be);
      if (!(b_be > 0.0)) {
        intFsq = rtInf * (rt_powd_snf(rtInf, 2.0 * varargin_2) *
                          std::exp(-2.0 * rt_powd_snf(rtInf, varargin_1)));
      } else {
        int32_T nsubs;
        boolean_T first_iteration;
        nsubs = ix - 2;
        for (k = 0; k <= nsubs; k++) {
          subs[k << 1] = interval[k];
          subs[(k << 1) + 1] = interval[k + 1];
        }
        q_ok = 0.0;
        err_ok = 0.0;
        first_iteration = true;
        emxInit_real_T(&x, 2, &hc_emlrtRTEI, true);
        emxInit_real_T(&b_x, 2, &nc_emlrtRTEI, true);
        emxInit_real_T(&xt, 2, &oc_emlrtRTEI, true);
        emxInit_real_T(&y, 2, &pc_emlrtRTEI, true);
        emxInit_real_T(&fx, 2, &qc_emlrtRTEI, true);
        emxInit_real_T(&b_dv1, 2, &rc_emlrtRTEI, true);
        emxInit_real_T(&b_y, 2, &rc_emlrtRTEI, true);
        int32_T exitg1;
        do {
          boolean_T guard1;
          exitg1 = 0;
          ix = x->size[0] * x->size[1];
          x->size[0] = 1;
          x->size[1] = 15 * (nsubs + 1);
          emxEnsureCapacity_real_T(x, ix, &hc_emlrtRTEI);
          ix = -1;
          for (k = 0; k <= nsubs; k++) {
            d = subs[k << 1];
            d1 = subs[(k << 1) + 1];
            midpt = (d + d1) / 2.0;
            halfh = (d1 - d) / 2.0;
            for (int32_T j{0}; j < 15; j++) {
              x->data[(ix + j) + 1] = dv[j] * halfh + midpt;
            }
            ix += 15;
          }
          ix = b_x->size[0] * b_x->size[1];
          b_x->size[0] = 1;
          b_x->size[1] = x->size[1];
          emxEnsureCapacity_real_T(b_x, ix, &ic_emlrtRTEI);
          ix = xt->size[0] * xt->size[1];
          xt->size[0] = 1;
          xt->size[1] = x->size[1];
          emxEnsureCapacity_real_T(xt, ix, &jc_emlrtRTEI);
          ix = x->size[1];
          for (k = 0; k < ix; k++) {
            d = x->data[k];
            midpt = d / (1.0 - d);
            b_x->data[k] = midpt * midpt;
            xt->data[k] = 2.0 * midpt / ((1.0 - d) * (1.0 - d));
          }
          guard1 = false;
          if (!first_iteration) {
            boolean_T exitg2;
            midpt = std::abs(b_x->data[0]);
            k = 0;
            exitg2 = false;
            while ((!exitg2) && (k <= b_x->size[1] - 2)) {
              halfh = midpt;
              midpt = std::abs(b_x->data[k + 1]);
              if (std::abs(b_x->data[k + 1] - b_x->data[k]) <=
                  2.2204460492503131E-14 * std::fmax(halfh, midpt)) {
                first_iteration = true;
                exitg2 = true;
              } else {
                k++;
              }
            }
            if (first_iteration) {
              int16_T dv_idx_1;
              dv_idx_1 = static_cast<int16_T>(x->size[1]);
              ix = fx->size[0] * fx->size[1];
              fx->size[0] = 1;
              fx->size[1] = x->size[1];
              emxEnsureCapacity_real_T(fx, ix, &lc_emlrtRTEI);
              for (ix = 0; ix < dv_idx_1; ix++) {
                fx->data[ix] = 0.0;
              }
            } else {
              guard1 = true;
            }
          } else {
            guard1 = true;
          }
          if (guard1) {
            first_iteration = false;
            ix = y->size[0] * y->size[1];
            y->size[0] = 1;
            y->size[1] = b_x->size[1];
            emxEnsureCapacity_real_T(y, ix, &m_emlrtRTEI);
            ix = b_x->size[1];
            for (k = 0; k < ix; k++) {
              y->data[k] = rt_powd_snf(b_x->data[k], varargin_1);
            }
            ix = b_dv1->size[0] * b_dv1->size[1];
            b_dv1->size[0] = 1;
            b_dv1->size[1] = y->size[1];
            emxEnsureCapacity_real_T(b_dv1, ix, &kc_emlrtRTEI);
            for (ix = 0; ix < y->size[1]; ix++) {
              b_dv1->data[ix] = -2.0 * y->data[ix];
            }
            ix = b_x->size[1];
            for (k = 0; k < ix; k++) {
              b_dv1->data[k] = std::exp(b_dv1->data[k]);
            }
            midpt = 2.0 * varargin_2;
            ix = b_y->size[0] * b_y->size[1];
            b_y->size[0] = 1;
            b_y->size[1] = b_x->size[1];
            emxEnsureCapacity_real_T(b_y, ix, &m_emlrtRTEI);
            ix = b_x->size[1];
            for (k = 0; k < ix; k++) {
              b_y->data[k] = rt_powd_snf(b_x->data[k], midpt);
            }
            if (b_y->size[1] == b_dv1->size[1]) {
              ix = fx->size[0] * fx->size[1];
              fx->size[0] = 1;
              fx->size[1] = b_y->size[1];
              emxEnsureCapacity_real_T(fx, ix, &mc_emlrtRTEI);
              for (ix = 0; ix < b_y->size[1]; ix++) {
                fx->data[ix] = b_y->data[ix] * b_dv1->data[ix] * xt->data[ix];
              }
            } else {
              binary_expand_op_4(fx, b_y, b_dv1, xt);
            }
          }
          if (first_iteration) {
            exitg1 = 1;
          } else {
            midpt = 0.0;
            ix = -1;
            for (k = 0; k <= nsubs; k++) {
              d = 0.0;
              d1 = 0.0;
              for (int32_T j{0}; j < 15; j++) {
                d += dv1[j] * fx->data[(ix + j) + 1];
                d1 += dv2[j] * fx->data[(ix + j) + 1];
              }
              ix += 15;
              halfh = (subs[(k << 1) + 1] - subs[k << 1]) / 2.0;
              d *= halfh;
              qsub[k] = d;
              midpt += d;
              errsub[k] = d1 * halfh;
            }
            intFsq = midpt + q_ok;
            tol = std::fmax(1.0E-10, 1.0E-6 * std::abs(intFsq));
            midpt = 2.0 * tol / b_be;
            halfh = 0.0;
            ix = 0;
            for (k = 0; k <= nsubs; k++) {
              d = errsub[k];
              abserrsubk = std::abs(d);
              if (abserrsubk <=
                  midpt * ((subs[(k << 1) + 1] - subs[k << 1]) / 2.0)) {
                err_ok += d;
                q_ok += qsub[k];
              } else {
                halfh += abserrsubk;
                ix++;
                subs[(ix - 1) << 1] = subs[k << 1];
                subs[((ix - 1) << 1) + 1] = subs[(k << 1) + 1];
              }
            }
            midpt = std::abs(err_ok) + halfh;
            if ((!std::isinf(intFsq)) && (!std::isnan(intFsq)) &&
                ((!std::isinf(midpt)) && (!std::isnan(midpt))) && (ix != 0) &&
                (!(midpt <= tol))) {
              nsubs = (ix << 1) - 1;
              if (nsubs + 1 > 650) {
                exitg1 = 1;
              } else {
                for (k = 0; k < ix; k++) {
                  subs[((((ix - k) << 1) - 1) << 1) + 1] =
                      subs[(((ix - k) - 1) << 1) + 1];
                  subs[(((ix - k) << 1) - 1) << 1] =
                      (subs[((ix - k) - 1) << 1] +
                       subs[(((ix - k) - 1) << 1) + 1]) /
                      2.0;
                  subs[((((ix - k) << 1) - 2) << 1) + 1] =
                      subs[(((ix - k) << 1) - 1) << 1];
                  subs[(((ix - k) << 1) - 2) << 1] = subs[((ix - k) - 1) << 1];
                }
              }
            } else {
              exitg1 = 1;
            }
          }
        } while (exitg1 == 0);
        emxFree_real_T(&b_y);
        emxFree_real_T(&b_dv1);
        emxFree_real_T(&fx);
        emxFree_real_T(&y);
        emxFree_real_T(&xt);
        emxFree_real_T(&b_x);
        emxFree_real_T(&x);
      }
      *sigmaT = std::sqrt(be * be * (quadgk(varargin_2, varargin_1) / intFsq));
    }
  } else if (wname[0] == 'a') {
    *cf = 6.0;
    *sigmaT = 1.4142135623730951;
  } else {
    *cf = 5.0;
    *sigmaT = 5.847705;
  }
  FourierFactor = 6.2831853071795862 / *cf;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  return FourierFactor;
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (wavCFandSD.cu)

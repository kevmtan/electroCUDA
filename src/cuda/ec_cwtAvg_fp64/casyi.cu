#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// casyi.cu
//
// Code generation for function 'casyi'
//

// Include files
#include "casyi.h"
#include "cospiAndSinpi.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_mexutil.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
int32_T casyi(const creal_T z, real_T fnu, int32_T kode, creal_T *y)
{
  real_T absxi;
  real_T absxr;
  real_T ak1_im;
  real_T ak1_re;
  real_T cz_im;
  real_T p1_re;
  real_T yr;
  int32_T nz;
  nz = 0;
  if (z.im == 0.0) {
    ak1_re = 0.15915494309189535 / z.re;
    ak1_im = 0.0;
  } else if (z.re == 0.0) {
    ak1_re = 0.0;
    ak1_im = -(0.15915494309189535 / z.im);
  } else {
    ak1_im = std::abs(z.re);
    absxi = std::abs(z.im);
    if (ak1_im > absxi) {
      absxi = z.im / z.re;
      absxr = z.re + absxi * z.im;
      ak1_re = (absxi * 0.0 + 0.15915494309189535) / absxr;
      ak1_im = (0.0 - absxi * 0.15915494309189535) / absxr;
    } else if (absxi == ak1_im) {
      if (z.re > 0.0) {
        absxi = 0.5;
      } else {
        absxi = -0.5;
      }
      if (z.im > 0.0) {
        absxr = 0.5;
      } else {
        absxr = -0.5;
      }
      ak1_re = (0.15915494309189535 * absxi + 0.0 * absxr) / ak1_im;
      ak1_im = (0.0 * absxi - 0.15915494309189535 * absxr) / ak1_im;
    } else {
      absxi = z.re / z.im;
      absxr = z.im + absxi * z.re;
      ak1_re = absxi * 0.15915494309189535 / absxr;
      ak1_im = (absxi * 0.0 - 0.15915494309189535) / absxr;
    }
  }
  if (ak1_im == 0.0) {
    if (ak1_re < 0.0) {
      yr = 0.0;
      absxr = std::sqrt(-ak1_re);
    } else {
      yr = std::sqrt(ak1_re);
      absxr = 0.0;
    }
  } else if (ak1_re == 0.0) {
    if (ak1_im < 0.0) {
      yr = std::sqrt(-ak1_im / 2.0);
      absxr = -yr;
    } else {
      yr = std::sqrt(ak1_im / 2.0);
      absxr = yr;
    }
  } else if (std::isnan(ak1_re)) {
    yr = rtNaN;
    absxr = rtNaN;
  } else if (std::isnan(ak1_im)) {
    yr = rtNaN;
    absxr = rtNaN;
  } else if (std::isinf(ak1_im)) {
    yr = std::abs(ak1_im);
    absxr = ak1_im;
  } else if (std::isinf(ak1_re)) {
    if (ak1_re < 0.0) {
      yr = 0.0;
      absxr = ak1_im * -ak1_re;
    } else {
      yr = ak1_re;
      absxr = 0.0;
    }
  } else {
    absxr = std::abs(ak1_re);
    absxi = std::abs(ak1_im);
    if ((absxr > 4.4942328371557893E+307) ||
        (absxi > 4.4942328371557893E+307)) {
      absxr *= 0.5;
      absxi = rt_hypotd_snf(absxr, absxi * 0.5);
      if (absxi > absxr) {
        yr = std::sqrt(absxi) * std::sqrt(absxr / absxi + 1.0);
      } else {
        yr = std::sqrt(absxi) * 1.4142135623730951;
      }
    } else {
      yr = std::sqrt((rt_hypotd_snf(absxr, absxi) + absxr) * 0.5);
    }
    if (ak1_re > 0.0) {
      absxr = 0.5 * (ak1_im / yr);
    } else {
      if (ak1_im < 0.0) {
        absxr = -yr;
      } else {
        absxr = yr;
      }
      yr = 0.5 * (ak1_im / absxr);
    }
  }
  if (kode == 2) {
    ak1_re = 0.0;
    cz_im = z.im;
    absxi = 0.0;
  } else {
    ak1_re = z.re;
    cz_im = z.im;
    absxi = z.re;
  }
  if (std::abs(absxi) > 700.92179369444591) {
    nz = -1;
    y->re = rtNaN;
    y->im = 0.0;
  } else {
    real_T aa;
    real_T aez;
    real_T ak;
    real_T b_re;
    real_T bb;
    real_T bk;
    real_T ck_im;
    real_T cs1_im;
    real_T cs1_re;
    real_T cs2_im;
    real_T cs2_re;
    real_T dk_im;
    real_T dk_re;
    real_T ez_im;
    real_T ez_re;
    real_T im;
    real_T re;
    real_T sqk;
    int32_T i;
    boolean_T errflag;
    boolean_T exitg1;
    absxi = fnu + fnu;
    if (ak1_re == 0.0) {
      ak1_re = std::cos(cz_im);
      cz_im = std::sin(cz_im);
    } else if (cz_im == 0.0) {
      ak1_re = std::exp(ak1_re);
      cz_im = 0.0;
    } else if (std::isinf(cz_im) && std::isinf(ak1_re) && (ak1_re < 0.0)) {
      ak1_re = 0.0;
      cz_im = 0.0;
    } else {
      ak1_im = std::exp(ak1_re / 2.0);
      ak1_re = ak1_im * (ak1_im * std::cos(cz_im));
      cz_im = ak1_im * (ak1_im * std::sin(cz_im));
    }
    re = yr * ak1_re - absxr * cz_im;
    im = yr * cz_im + absxr * ak1_re;
    absxr = 0.0;
    if (absxi > 4.7170688552396617E-153) {
      absxr = absxi * absxi;
    }
    ez_re = 8.0 * z.re;
    ez_im = 8.0 * z.im;
    aez = 8.0 * rt_hypotd_snf(z.re, z.im);
    if (z.im != 0.0) {
      bk = internal::scalar::cospiAndSinpi(
          fnu - static_cast<real_T>(static_cast<int32_T>(fnu)), &p1_re);
      if (z.im < 0.0) {
        bk = -bk;
      }
      if (static_cast<int32_T>(fnu) != 0) {
        bk = -bk;
      } else {
        p1_re = -p1_re;
      }
    } else {
      p1_re = 0.0;
      bk = 0.0;
    }
    sqk = absxr - 1.0;
    yr = 2.2204460492503131E-16 / aez * std::abs(absxr - 1.0);
    ak1_re = 1.0;
    cs1_re = 1.0;
    cs1_im = 0.0;
    cs2_re = 1.0;
    cs2_im = 0.0;
    cz_im = 1.0;
    ck_im = 0.0;
    ak = 0.0;
    aa = 1.0;
    bb = aez;
    dk_re = ez_re;
    dk_im = ez_im;
    errflag = true;
    i = 1;
    exitg1 = false;
    while ((!exitg1) && (i < 46)) {
      cz_im *= sqk;
      ck_im *= sqk;
      if (dk_im == 0.0) {
        if (ck_im == 0.0) {
          b_re = cz_im / dk_re;
          ck_im = 0.0;
        } else if (cz_im == 0.0) {
          b_re = 0.0;
          ck_im /= dk_re;
        } else {
          b_re = cz_im / dk_re;
          ck_im /= dk_re;
        }
      } else if (dk_re == 0.0) {
        if (cz_im == 0.0) {
          b_re = ck_im / dk_im;
          ck_im = 0.0;
        } else if (ck_im == 0.0) {
          b_re = 0.0;
          ck_im = -(cz_im / dk_im);
        } else {
          b_re = ck_im / dk_im;
          ck_im = -(cz_im / dk_im);
        }
      } else {
        ak1_im = std::abs(dk_re);
        absxi = std::abs(dk_im);
        if (ak1_im > absxi) {
          absxi = dk_im / dk_re;
          absxr = dk_re + absxi * dk_im;
          b_re = (cz_im + absxi * ck_im) / absxr;
          ck_im = (ck_im - absxi * cz_im) / absxr;
        } else if (absxi == ak1_im) {
          if (dk_re > 0.0) {
            absxi = 0.5;
          } else {
            absxi = -0.5;
          }
          if (dk_im > 0.0) {
            absxr = 0.5;
          } else {
            absxr = -0.5;
          }
          b_re = (cz_im * absxi + ck_im * absxr) / ak1_im;
          ck_im = (ck_im * absxi - cz_im * absxr) / ak1_im;
        } else {
          absxi = dk_re / dk_im;
          absxr = dk_im + absxi * dk_re;
          b_re = (absxi * cz_im + ck_im) / absxr;
          ck_im = (absxi * ck_im - cz_im) / absxr;
        }
      }
      cz_im = b_re;
      cs2_re += b_re;
      cs2_im += ck_im;
      ak1_re = -ak1_re;
      cs1_re += b_re * ak1_re;
      cs1_im += ck_im * ak1_re;
      dk_re += ez_re;
      dk_im += ez_im;
      aa = aa * std::abs(sqk) / bb;
      bb += aez;
      ak += 8.0;
      sqk -= ak;
      if (aa <= yr) {
        errflag = false;
        exitg1 = true;
      } else {
        i++;
      }
    }
    if (errflag) {
      nz = -2;
    } else {
      if (z.re + z.re < 700.92179369444591) {
        ak1_re = -2.0 * z.re;
        cz_im = -2.0 * z.im;
        if (ak1_re == 0.0) {
          ak1_re = std::cos(cz_im);
          cz_im = std::sin(cz_im);
        } else if (cz_im == 0.0) {
          ak1_re = std::exp(ak1_re);
          cz_im = 0.0;
        } else if (std::isinf(cz_im) && std::isinf(ak1_re) && (ak1_re < 0.0)) {
          ak1_re = 0.0;
          cz_im = 0.0;
        } else {
          ak1_im = std::exp(ak1_re / 2.0);
          ak1_re = ak1_im * (ak1_im * std::cos(cz_im));
          cz_im = ak1_im * (ak1_im * std::sin(cz_im));
        }
        b_re = ak1_re * cs2_re - cz_im * cs2_im;
        absxi = ak1_re * cs2_im + cz_im * cs2_re;
        cs1_re += b_re * p1_re - absxi * bk;
        cs1_im += b_re * bk + absxi * p1_re;
      }
      y->re = cs1_re * re - cs1_im * im;
      y->im = cs1_re * im + cs1_im * re;
    }
  }
  return nz;
}

} // namespace coder

// End of code generation (casyi.cu)

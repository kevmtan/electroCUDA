#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_cwtAvg.cu
//
// Code generation for function 'ec_cwtAvg'
//

// Include files
#include "ec_cwtAvg.h"
#include "cwtfilterbank.h"
#include "div.h"
#include "ec_cwtAvg_data.h"
#include "ec_cwtAvg_emxutil.h"
#include "ec_cwtAvg_mexutil.h"
#include "ec_cwtAvg_types.h"
#include "numCpsi.h"
#include "resample.h"
#include "rt_nonfinite.h"
#include "scNormalize.h"
#include "vvarstd.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hipfft/hipfft.h"
#include "hip/hip_math_constants.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    26,                                                    // lineNo
    1,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    1,                                                     // lineNo
    22,                                                    // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo
    c_emlrtRTEI{
        105,      // lineNo
        20,       // colNo
        "varstd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
        "varstd.m" // pName
    };

static emlrtRTEInfo d_emlrtRTEI{
    135,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    143,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "cuFFTNDCallback", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "cuFFTNDCallback.p" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    138,  // lineNo
    19,   // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    138,  // lineNo
    6,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    236,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    35,                                                    // lineNo
    28,                                                    // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    298,             // lineNo
    18,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    297,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    35,                                                    // lineNo
    5,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    41,                                                    // lineNo
    9,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    39,                                                    // lineNo
    9,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    24,                                                    // lineNo
    1,                                                     // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    287,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    151,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    293,             // lineNo
    14,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    39,                                                    // lineNo
    17,                                                    // colNo
    "ec_cwtAvg",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwtAvg.m" // pName
};

// Function Declarations
static __global__ void ec_cwtAvg_kernel1(const int32_T ch,
                                         const emxArray_real_T x,
                                         const int32_T b, emxArray_real_T xv,
                                         emxArray_real_T b_x, int32_T x_dim0);

static __global__ void ec_cwtAvg_kernel10(const real_T cpsi,
                                          emxArray_real_T savgpTMP);

static __global__ void ec_cwtAvg_kernel11(const emxArray_real_T savgpTMP,
                                          const int32_T b_savgpTMP,
                                          emxArray_real_T xCh);

static __global__ void ec_cwtAvg_kernel12(const int32_T abswt2S,
                                          emxArray_real_T savgpTMP);

static __global__ void ec_cwtAvg_kernel13(const int32_T abswt2S,
                                          emxArray_real_T savgpTMP);

static __global__ void ec_cwtAvg_kernel14(const real_T cpsi, emxArray_real_T c);

static __global__ void ec_cwtAvg_kernel15(const real_T cpsi, const int32_T i2,
                                          emxArray_real_T c);

static __global__ void ec_cwtAvg_kernel16(const real_T cfsposdft_re,
                                          const emxArray_creal_T cfsposdft,
                                          const int32_T b_cfsposdft,
                                          emxArray_creal_T cfs);

static __global__ void ec_cwtAvg_kernel17(const emxArray_real_T xv,
                                          const int32_T b_xv,
                                          emxArray_creal_T xposdft);

static __global__ void ec_cwtAvg_kernel18(const int32_T b,
                                          emxArray_creal_T xposdft,
                                          uint32_T xSize_dim1);

static __global__ void ec_cwtAvg_kernel2(const int32_T ch,
                                         const emxArray_real_T x,
                                         const int32_T b, emxArray_real_T b_x,
                                         int32_T x_dim0);

static __global__ void ec_cwtAvg_kernel3(const int32_T i2, const int32_T b,
                                         emxArray_real_T x);

static __global__ void ec_cwtAvg_kernel4(const emxArray_real_T x,
                                         const int32_T b, emxArray_real_T xv);

static __global__ void ec_cwtAvg_kernel5(const int32_T xSize,
                                         emxArray_creal_T cfs);

static __global__ void ec_cwtAvg_kernel6(const int32_T ch,
                                         const emxArray_real_T x,
                                         const int32_T b_x, emxArray_real_T c_x,
                                         int32_T x_dim0);

static __global__ void ec_cwtAvg_kernel7(const emxArray_creal_T cfs,
                                         const int32_T b, emxArray_real_T y);

static __global__ void ec_cwtAvg_kernel8(const emxArray_real_T y,
                                         const int32_T b, emxArray_real_T b_y);

static __global__ void ec_cwtAvg_kernel9(const emxArray_real_T abswt2S,
                                         const int32_T b,
                                         const int32_T b_abswt2S,
                                         emxArray_real_T y, int32_T y_dim0,
                                         int32_T abswt2S_dim0);

static void gpuEmxEnsureCapacity_creal_T(const emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu,
                                         boolean_T needsCopy);

static void gpuEmxFree_creal_T(emxArray_creal_T *gpu);

static void gpuEmxMemcpyCpuToGpu_creal_T(emxArray_creal_T *gpu,
                                         const emxArray_creal_T *cpu);

static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu);

static void gpuEmxReset_creal_T(emxArray_creal_T *gpu);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel1(
    const int32_T ch, const emxArray_real_T x, const int32_T b,
    emxArray_real_T xv, emxArray_real_T b_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T xtmp;
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xtmp = x.data[j2 + x_dim0 * ch];
    b_x.data[j2] = xtmp;
    xv.data[j2] = xtmp;
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_cwtAvg_kernel10(const real_T cpsi,
                                                     emxArray_real_T savgpTMP)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    savgpTMP.data[0] = cpsi;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel11(
    const emxArray_real_T savgpTMP, const int32_T b_savgpTMP,
    emxArray_real_T xCh)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_savgpTMP);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xCh.data[j2] = savgpTMP.data[j2];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_cwtAvg_kernel12(const int32_T abswt2S,
                                                       emxArray_real_T savgpTMP)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(abswt2S);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    savgpTMP.data[j2] = 0.0;
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_cwtAvg_kernel13(const int32_T abswt2S,
                                                       emxArray_real_T savgpTMP)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(abswt2S);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    savgpTMP.data[j2] = HIP_NAN;
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_cwtAvg_kernel14(const real_T cpsi,
                                                     emxArray_real_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = cpsi;
  }
}

static __global__ __launch_bounds__(32, 1) void ec_cwtAvg_kernel15(
    const real_T cpsi, const int32_T i2, emxArray_real_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[i2 - 1] = cpsi;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel16(
    const real_T cfsposdft_re, const emxArray_creal_T cfsposdft,
    const int32_T b_cfsposdft, emxArray_creal_T cfs)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_cfsposdft);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T ai;
    real_T xtmp;
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xtmp = cfsposdft.data[j2].re;
    ai = cfsposdft.data[j2].im;
    if (ai == 0.0) {
      cfs.data[j2].re = xtmp / cfsposdft_re;
      cfs.data[j2].im = 0.0;
    } else if (xtmp == 0.0) {
      cfs.data[j2].re = 0.0;
      cfs.data[j2].im = ai / cfsposdft_re;
    } else {
      cfs.data[j2].re = xtmp / cfsposdft_re;
      cfs.data[j2].im = ai / cfsposdft_re;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel17(
    const emxArray_real_T xv, const int32_T b_xv, emxArray_creal_T xposdft)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_xv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xposdft.data[j2].re = xv.data[j2];
    xposdft.data[j2].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel18(
    const int32_T b, emxArray_creal_T xposdft, uint32_T xSize_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    b_j1 = static_cast<int32_T>(idx);
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].re =
        xposdft.data[b_j1 + 1].re;
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].im =
        -xposdft.data[b_j1 + 1].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel2(
    const int32_T ch, const emxArray_real_T x, const int32_T b,
    emxArray_real_T b_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    b_x.data[j2] = x.data[j2 + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void ec_cwtAvg_kernel3(const int32_T i2,
                                                              const int32_T b,
                                                              emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T xtmp;
    int32_T b_j1;
    int32_T j2;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i2 - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel4(
    const emxArray_real_T x, const int32_T b, emxArray_real_T xv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    xv.data[j2] = x.data[j2];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_cwtAvg_kernel5(const int32_T xSize,
                                                      emxArray_creal_T cfs)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(xSize);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    cfs.data[j2].re = 0.0;
    cfs.data[j2].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel6(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T c_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T j2;
    j2 = static_cast<int32_T>(idx);
    c_x.data[j2] = x.data[j2 + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel7(
    const emxArray_creal_T cfs, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    b_j1 = static_cast<int32_T>(idx);
    y.data[b_j1] = hypot(cfs.data[b_j1].re, cfs.data[b_j1].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel8(
    const emxArray_real_T y, const int32_T b, emxArray_real_T b_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    b_j1 = static_cast<int32_T>(idx);
    b_y.data[b_j1] = y.data[b_j1] * y.data[b_j1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwtAvg_kernel9(
    const emxArray_real_T abswt2S, const int32_T b, const int32_T b_abswt2S,
    emxArray_real_T y, int32_T y_dim0, int32_T abswt2S_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_abswt2S) + 1UL) *
                (static_cast<uint64_T>(b) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    int32_T j2;
    b_j1 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    j2 = static_cast<int32_T>((idx - static_cast<uint64_T>(b_j1)) /
                              (static_cast<uint64_T>(b) + 1UL));
    y.data[b_j1 + y_dim0 * j2] = abswt2S.data[b_j1 + abswt2S_dim0 * j2];
  }
}

static void gpuEmxEnsureCapacity_creal_T(const emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu,
                                         boolean_T needsCopy)
{
  creal_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(creal_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(creal_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxFree_creal_T(emxArray_creal_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_creal_T(emxArray_creal_T *gpu,
                                         const emxArray_creal_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(creal_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(creal_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

static void gpuEmxReset_creal_T(emxArray_creal_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_creal_T));
}

//
// function [y,freqs] = ec_cwtAvg(x,fs,fLims,fVoices,ds)
void ec_cwtAvg(const emxArray_real_T *cpu_x, real_T fs, const real_T fLims[2],
               real_T fVoices, const real_T ds[2], emxArray_cell_wrap_0 *y,
               emxArray_real_T *freqs)
{
  coder::cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal_T gpu_cfs;
  emxArray_creal_T gpu_cfsposdft;
  emxArray_creal_T gpu_xposdft;
  emxArray_creal_T *cpu_cfs;
  emxArray_creal_T *cpu_cfsposdft;
  emxArray_creal_T *cpu_xposdft;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_x;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_x;
  emxArray_real_T gpu_abswt2S;
  emxArray_real_T gpu_c;
  emxArray_real_T gpu_r;
  emxArray_real_T gpu_savgpTMP;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xCh;
  emxArray_real_T gpu_xv;
  emxArray_real_T gpu_y;
  emxArray_real_T *b_cpu_x;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_x;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_abswt2S;
  emxArray_real_T *cpu_c;
  emxArray_real_T *cpu_r;
  emxArray_real_T *cpu_savgpTMP;
  emxArray_real_T *cpu_xCh;
  emxArray_real_T *cpu_xv;
  emxArray_real_T *cpu_y;
  emxArray_real_T *d_cpu_x;
  int32_T bcoef;
  int32_T i1;
  boolean_T doDownsample;
  boolean_T r_outdatedOnCpu;
  boolean_T r_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T xv_outdatedOnCpu;
  gpuEmxReset_real_T(&d_gpu_x);
  gpuEmxReset_real_T(&b_gpu_x);
  gpuEmxReset_real_T(&gpu_r);
  gpuEmxReset_real_T(&gpu_xCh);
  gpuEmxReset_real_T(&gpu_c);
  gpuEmxReset_real_T(&c_gpu_y);
  gpuEmxReset_real_T(&gpu_abswt2S);
  gpuEmxReset_real_T(&b_gpu_y);
  gpuEmxReset_real_T(&gpu_y);
  gpuEmxReset_creal_T(&gpu_cfs);
  gpuEmxReset_real_T(&c_gpu_x);
  gpuEmxReset_creal_T(&gpu_cfsposdft);
  gpuEmxReset_creal_T(&gpu_xposdft);
  gpuEmxReset_real_T(&gpu_xv);
  gpuEmxReset_real_T(&gpu_savgpTMP);
  gpuEmxReset_real_T(&gpu_x);
  x_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInitStruct_cwtfilterbank(&fb, &v_emlrtRTEI, true);
  //  CWT scale spectrum - CUDA binary wrapper (double-precision, FP64)
  //  CWT uses morse wavelets, as they account for unequal variance-covariance
  //  across freqs. L1-norm is applied to mitigate 1/f decay of neuronal field
  //  potentials. Log-transform & robust z-scoring converts absolute valued CWT
  //  outputs into normal distribution.
  //
  //  Kevin Tan, 2022 (kevmtan.github.io)
  //  Input validation
  // 'ec_cwtAvg:11' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_cwtAvg:12' fs (1,1) double = 1000
  //  Sampling rate
  // 'ec_cwtAvg:13' fLims (1,2) double = [1 300]
  //  Frequency limits
  // 'ec_cwtAvg:14' fVoices (1,1) double = 10
  //  Voices per octave
  // 'ec_cwtAvg:15' ds (1,2) double = [1 1]
  //  Frequency limits
  // 'ec_cwtAvg:17' coder.gpu.kernelfun;
  // 'ec_cwtAvg:18' fVoices = round(fVoices);
  fVoices = std::round(fVoices);
  // 'ec_cwtAvg:19' nFrames = height(x);
  // 'ec_cwtAvg:20' nChs = width(x);
  // 'ec_cwtAvg:21' doDownsample = ds(2)>ds(1);
  doDownsample = (ds[1] > ds[0]);
  //  Prep CWT
  // 'ec_cwtAvg:24' fb = cwtfilterbank(Wavelet="Morse",SignalLength=nFrames,...
  // 'ec_cwtAvg:25'
  // SamplingFrequency=fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
  coder::cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(cpu_x->size[0]),
                                     fs, fLims, fVoices);
  // 'ec_cwtAvg:26' freqs = centerFrequencies(fb);
  bcoef = freqs->size[0];
  freqs->size[0] = fb.WaveletCenterFrequencies->size[0];
  emxEnsureCapacity_real_T(freqs, bcoef, &emlrtRTEI);
  for (bcoef = 0; bcoef < fb.WaveletCenterFrequencies->size[0]; bcoef++) {
    freqs->data[bcoef] = fb.WaveletCenterFrequencies->data[bcoef];
  }
  //  Preallocate
  // 'ec_cwtAvg:29' tmp = cell(1,nChs);
  bcoef = y->size[0] * y->size[1];
  y->size[0] = 1;
  y->size[1] = cpu_x->size[1];
  emxEnsureCapacity_cell_wrap_0(y, bcoef, &b_emlrtRTEI);
  // 'ec_cwtAvg:30' y = coder.nullcopy(tmp);
  //  y = coder.nullcopy(x); % Preallocate output
  //  CWT average power (L1-norm & variance-norm usng weighted integrals)
  // 'ec_cwtAvg:34' for ch = 1:nChs
  i1 = cpu_x->size[1];
  emxInit_real_T(&cpu_savgpTMP, 2, &w_emlrtRTEI, true);
  emxInit_real_T(&cpu_xv, 2, &d_emlrtRTEI, true);
  emxInit_creal_T(&cpu_xposdft, 2, &e_emlrtRTEI, true);
  emxInit_creal_T(&cpu_cfsposdft, 2, &x_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_x, 2, &g_emlrtRTEI, true);
  emxInit_creal_T(&cpu_cfs, 2, &j_emlrtRTEI, true);
  emxInit_real_T(&cpu_y, 2, &y_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_y, 2, &y_emlrtRTEI, true);
  emxInit_real_T(&cpu_abswt2S, 2, &n_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_y, 2, &o_emlrtRTEI, true);
  emxInit_real_T(&cpu_c, 1, &r_emlrtRTEI, true);
  emxInit_real_T(&cpu_xCh, 1, &s_emlrtRTEI, true);
  emxInit_real_T(&cpu_r, 2, &ab_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_x, 1, &c_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_x, 1, &k_emlrtRTEI, true);
  for (int32_T ch{0}; ch < i1; ch++) {
    real_T alpha1;
    real_T cpsi;
    int32_T acoef;
    int32_T b_bcoef;
    int32_T i2;
    int32_T i5;
    int32_T j2;
    int32_T nd2;
    uint32_T xSize[2];
    uint32_T defaultSL_idx_1;
    boolean_T validLaunchParams;
    boolean_T xCh_outdatedOnCpu;
    // 'ec_cwtAvg:35' xCh = scaleSpectrum(fb,x(:,ch),SpectrumType="density")';
    defaultSL_idx_1 = static_cast<uint32_T>(fb.Scales->size[1]);
    nd2 = cpu_x->size[0];
    bcoef = c_cpu_x->size[0];
    c_cpu_x->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real_T(c_cpu_x, bcoef, &c_emlrtRTEI);
    bcoef = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    cpu_xv->size[1] = cpu_x->size[0];
    emxEnsureCapacity_real_T(cpu_xv, bcoef, &d_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nd2 - 1), &grid, &block,
                            2147483647U);
    if (x_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real_T(cpu_xv, &gpu_xv, true);
    r_outdatedOnCpu = false;
    gpuEmxEnsureCapacity_real_T(c_cpu_x, &b_gpu_x, true);
    if (x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel1<<<grid, block>>>(ch, gpu_x, nd2 - 1, gpu_xv, b_gpu_x,
                                         cpu_x->size[0U]);
    }
    r_outdatedOnGpu = false;
    gpuEmxMemcpyGpuToCpu_real_T(c_cpu_x, &b_gpu_x);
    fb.sigvar = coder::vvarstd(c_cpu_x, cpu_x->size[0]);
    if (fb.SignalPad > 0.0) {
      cpsi = fb.SignalPad;
      if (cpsi < 1.0) {
        i2 = 0;
      } else {
        i2 = static_cast<int32_T>(cpsi);
      }
      cpsi = (static_cast<real_T>(cpu_x->size[0]) - fb.SignalPad) + 1.0;
      if (cpsi > cpu_x->size[0]) {
        j2 = 0;
        b_bcoef = 1;
        i5 = -1;
      } else {
        j2 = cpu_x->size[0] - 1;
        b_bcoef = -1;
        i5 = static_cast<int32_T>(cpsi) - 1;
      }
      bcoef = b_cpu_x->size[0] * b_cpu_x->size[1];
      b_cpu_x->size[0] = 1;
      b_cpu_x->size[1] = i2;
      emxEnsureCapacity_real_T(b_cpu_x, bcoef, &g_emlrtRTEI);
      mwGetLaunchParameters1D(computeNumIters(i2 - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(b_cpu_x, &c_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel2<<<grid, block>>>(ch, gpu_x, i2 - 1, c_gpu_x,
                                           cpu_x->size[0U]);
      }
      mwGetLaunchParameters1D(computeNumIters((i2 >> 1) - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel3<<<grid, block>>>(i2, (i2 >> 1) - 1, c_gpu_x);
      }
      bcoef = cpu_xv->size[0] * cpu_xv->size[1];
      cpu_xv->size[0] = 1;
      cpu_xv->size[1] =
          ((cpu_x->size[0] + b_cpu_x->size[1]) + div_s32(i5 - j2, b_bcoef)) + 1;
      emxEnsureCapacity_real_T(cpu_xv, bcoef, &i_emlrtRTEI);
      acoef = b_cpu_x->size[1];
      mwGetLaunchParameters1D(computeNumIters(acoef - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_xv, &gpu_xv, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel4<<<grid, block>>>(c_gpu_x, acoef - 1, gpu_xv);
      }
      xv_outdatedOnCpu = true;
      acoef = cpu_x->size[0];
      for (bcoef = 0; bcoef < acoef; bcoef++) {
        if (xv_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_xv, &gpu_xv);
        }
        cpu_xv->data[bcoef + b_cpu_x->size[1]] =
            cpu_x->data[bcoef + cpu_x->size[0] * ch];
        xv_outdatedOnCpu = false;
        r_outdatedOnGpu = true;
        x_needsGpuEnsureCapacity = true;
        r_outdatedOnCpu = true;
      }
      acoef = div_s32(i5 - j2, b_bcoef);
      for (bcoef = 0; bcoef <= acoef; bcoef++) {
        if (xv_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_xv, &gpu_xv);
        }
        cpu_xv->data[(bcoef + cpu_x->size[0]) + b_cpu_x->size[1]] =
            cpu_x->data[(j2 + b_bcoef * bcoef) + cpu_x->size[0] * ch];
        xv_outdatedOnCpu = false;
        r_outdatedOnGpu = true;
        x_needsGpuEnsureCapacity = true;
        r_outdatedOnCpu = true;
      }
    }
    if (cpu_xv->size[1] == 0) {
      r_outdatedOnCpu = false;
      cpu_xposdft->size[0] = 1;
      cpu_xposdft->size[1] = 0;
    } else if (cpu_xv->size[1] -
                   (static_cast<int32_T>(
                        static_cast<uint32_T>(cpu_xv->size[1]) >> 1)
                    << 1) ==
               1) {
      hipfftHandle fftPlanHandle;
      bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
      cpu_xposdft->size[0] = 1;
      cpu_xposdft->size[1] = cpu_xv->size[1];
      emxEnsureCapacity_creal_T(cpu_xposdft, bcoef, &e_emlrtRTEI);
      j2 = cpu_xv->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
      if (r_outdatedOnCpu) {
        gpuEmxEnsureCapacity_real_T(cpu_xv, &gpu_xv, !r_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_creal_T(cpu_xposdft, &gpu_xposdft, true);
      if (r_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_xv, cpu_xv);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel17<<<grid, block>>>(gpu_xv, j2, gpu_xposdft);
      }
      nd2 = cpu_xposdft->size[1];
      fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_Z2Z, 1);
      hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&gpu_xposdft.data[0],
                   (hipfftDoubleComplex *)&gpu_xposdft.data[0], HIPFFT_FORWARD);
      r_outdatedOnCpu = true;
    } else {
      hipfftHandle b_fftPlanHandle;
      nd2 = cpu_xv->size[1];
      for (j2 = 0; j2 < 2; j2++) {
        xSize[j2] = static_cast<uint32_T>(cpu_xv->size[j2]);
      }
      bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
      cpu_xposdft->size[0] = 1;
      cpu_xposdft->size[1] = static_cast<int32_T>(xSize[1]);
      emxEnsureCapacity_creal_T(cpu_xposdft, bcoef, &f_emlrtRTEI);
      b_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_D2Z, 1);
      if (r_outdatedOnCpu) {
        gpuEmxEnsureCapacity_real_T(cpu_xv, &gpu_xv, !r_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_creal_T(cpu_xposdft, &gpu_xposdft, true);
      if (r_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_xv, cpu_xv);
      }
      hipfftExecD2Z(b_fftPlanHandle, (hipfftDoubleReal *)&gpu_xv.data[0],
                   (hipfftDoubleComplex *)&gpu_xposdft.data[0]);
      mwGetLaunchParameters1D(
          computeNumIters(
              static_cast<int32_T>(
                  static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) /
                  2.0) -
              2),
          &grid, &block, 2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel18<<<grid, block>>>(
            static_cast<int32_T>(
                static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) / 2.0) -
                2,
            gpu_xposdft, xSize[1]);
      }
      r_outdatedOnCpu = true;
    }
    bcoef = fb.PsiDFT->size[0];
    nd2 = fb.PsiDFT->size[1];
    i2 = cpu_xposdft->size[1];
    if (nd2 <= i2) {
      i2 = nd2;
    }
    if (fb.PsiDFT->size[1] == 1) {
      b_bcoef = cpu_xposdft->size[1];
    } else if (cpu_xposdft->size[1] == 1) {
      b_bcoef = fb.PsiDFT->size[1];
    } else if (cpu_xposdft->size[1] == fb.PsiDFT->size[1]) {
      b_bcoef = cpu_xposdft->size[1];
    } else {
      b_bcoef = i2;
    }
    r_outdatedOnGpu = false;
    nd2 = cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
    cpu_cfsposdft->size[0] = bcoef;
    cpu_cfsposdft->size[1] = b_bcoef;
    emxEnsureCapacity_creal_T(cpu_cfsposdft, nd2, &h_emlrtRTEI);
    if ((bcoef != 0) && (b_bcoef != 0)) {
      acoef = (cpu_xposdft->size[1] != 1);
      bcoef = (fb.PsiDFT->size[1] != 1);
      j2 = b_bcoef - 1;
      for (int32_T k{0}; k <= j2; k++) {
        nd2 = acoef * k;
        i2 = bcoef * k;
        b_bcoef = (fb.PsiDFT->size[0] != 1);
        i5 = cpu_cfsposdft->size[0] - 1;
        for (int32_T b_k{0}; b_k <= i5; b_k++) {
          if (r_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_creal_T(cpu_xposdft, &gpu_xposdft);
          }
          cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].re =
              fb.PsiDFT->data[b_bcoef * b_k + fb.PsiDFT->size[0] * i2] *
              cpu_xposdft->data[nd2].re;
          r_outdatedOnCpu = false;
          cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].im =
              fb.PsiDFT->data[b_bcoef * b_k + fb.PsiDFT->size[0] * i2] *
              cpu_xposdft->data[nd2].im;
          r_outdatedOnGpu = true;
        }
      }
    }
    if ((cpu_cfsposdft->size[0] == 0) || (cpu_cfsposdft->size[1] == 0)) {
      for (j2 = 0; j2 < 2; j2++) {
        xSize[j2] = static_cast<uint32_T>(cpu_cfsposdft->size[j2]);
      }
      xSize[1] = static_cast<uint32_T>(cpu_cfsposdft->size[1]);
      bcoef = cpu_cfs->size[0] * cpu_cfs->size[1];
      cpu_cfs->size[0] = static_cast<int32_T>(xSize[0]);
      cpu_cfs->size[1] = cpu_cfsposdft->size[1];
      emxEnsureCapacity_creal_T(cpu_cfs, bcoef, &j_emlrtRTEI);
      j2 = static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1;
      mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal_T(cpu_cfs, &gpu_cfs, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel5<<<grid, block>>>(j2, gpu_cfs);
      }
      r_outdatedOnCpu = true;
    } else {
      hipfftHandle c_fftPlanHandle;
      nd2 = cpu_cfsposdft->size[1];
      c_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, cpu_cfsposdft->size[0],
                                         1, HIPFFT_Z2Z, cpu_cfsposdft->size[0]);
      gpuEmxEnsureCapacity_creal_T(cpu_cfsposdft, &gpu_cfsposdft,
                                   !r_outdatedOnGpu);
      if (r_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_cfsposdft, cpu_cfsposdft);
      }
      hipfftExecZ2Z(c_fftPlanHandle,
                   (hipfftDoubleComplex *)&gpu_cfsposdft.data[0],
                   (hipfftDoubleComplex *)&gpu_cfsposdft.data[0], HIPFFT_BACKWARD);
      bcoef = cpu_cfs->size[0] * cpu_cfs->size[1];
      cpu_cfs->size[0] = cpu_cfsposdft->size[0];
      cpu_cfs->size[1] = cpu_cfsposdft->size[1];
      emxEnsureCapacity_creal_T(cpu_cfs, bcoef, &j_emlrtRTEI);
      j2 = cpu_cfsposdft->size[1];
      i5 = cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(i5), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal_T(cpu_cfs, &gpu_cfs, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel16<<<grid, block>>>(static_cast<real_T>(j2),
                                            gpu_cfsposdft, i5, gpu_cfs);
      }
      r_outdatedOnCpu = true;
    }
    if (fb.SignalPad > 0.0) {
      cpsi = fb.SignalPad + 1.0;
      alpha1 = fb.SignalPad + fb.SignalLength;
      if (cpsi > alpha1) {
        nd2 = 0;
        i2 = 0;
      } else {
        nd2 = static_cast<int32_T>(cpsi) - 1;
        i2 = static_cast<int32_T>(alpha1);
      }
      j2 = cpu_cfs->size[0];
      acoef = i2 - nd2;
      for (bcoef = 0; bcoef < acoef; bcoef++) {
        for (b_bcoef = 0; b_bcoef < j2; b_bcoef++) {
          if (r_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_creal_T(cpu_cfs, &gpu_cfs);
          }
          cpu_cfs->data[b_bcoef + j2 * bcoef] =
              cpu_cfs->data[b_bcoef + cpu_cfs->size[0] * (nd2 + bcoef)];
          r_outdatedOnCpu = false;
        }
      }
      bcoef = cpu_cfs->size[0] * cpu_cfs->size[1];
      cpu_cfs->size[1] = i2 - nd2;
      emxEnsureCapacity_creal_T(cpu_cfs, bcoef, &j_emlrtRTEI);
    }
    cpsi =
        coder::wavelet::internal::cwt::numCpsi(fb.Wavelet, fb.Gamma, fb.Beta);
    bcoef = d_cpu_x->size[0];
    d_cpu_x->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real_T(d_cpu_x, bcoef, &k_emlrtRTEI);
    j2 = cpu_x->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
    if (x_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, true);
    }
    x_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real_T(d_cpu_x, &d_gpu_x, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel6<<<grid, block>>>(ch, gpu_x, j2, d_gpu_x,
                                         cpu_x->size[0U]);
    }
    if (r_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_creal_T(cpu_cfs, &gpu_cfs);
    }
    gpuEmxMemcpyGpuToCpu_real_T(d_cpu_x, &d_gpu_x);
    coder::wavelet::internal::cwt::scNormalize(
        cpu_cfs, cpsi, fb.Scales, coder::vvarstd(d_cpu_x, cpu_x->size[0]));
    nd2 = cpu_cfs->size[0] * cpu_cfs->size[1];
    for (j2 = 0; j2 < 2; j2++) {
      xSize[j2] = static_cast<uint32_T>(cpu_cfs->size[j2]);
    }
    bcoef = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = static_cast<int32_T>(xSize[0]);
    cpu_y->size[1] = static_cast<int32_T>(xSize[1]);
    emxEnsureCapacity_real_T(cpu_y, bcoef, &l_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nd2 - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_creal_T(cpu_cfs, &gpu_cfs, false);
    gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, true);
    gpuEmxMemcpyCpuToGpu_creal_T(&gpu_cfs, cpu_cfs);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel7<<<grid, block>>>(gpu_cfs, nd2 - 1, gpu_y);
    }
    for (j2 = 0; j2 < 2; j2++) {
      xSize[j2] = static_cast<uint32_T>(cpu_y->size[j2]);
    }
    bcoef = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = static_cast<int32_T>(xSize[0]);
    b_cpu_y->size[1] = static_cast<int32_T>(xSize[1]);
    emxEnsureCapacity_real_T(b_cpu_y, bcoef, &m_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(xSize[0]) *
                                                static_cast<int32_T>(xSize[1]) -
                                            1),
                            &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel8<<<grid, block>>>(
          gpu_y,
          static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1,
          b_gpu_y);
    }
    r_outdatedOnCpu = true;
    bcoef = fb.Scales->size[1];
    if (b_cpu_y->size[0] == bcoef) {
      r_outdatedOnGpu = false;
      bcoef = cpu_abswt2S->size[0] * cpu_abswt2S->size[1];
      cpu_abswt2S->size[0] = b_cpu_y->size[0];
      cpu_abswt2S->size[1] = b_cpu_y->size[1];
      emxEnsureCapacity_real_T(cpu_abswt2S, bcoef, &n_emlrtRTEI);
      for (bcoef = 0; bcoef < b_cpu_y->size[1]; bcoef++) {
        for (b_bcoef = 0; b_bcoef < b_cpu_y->size[0]; b_bcoef++) {
          if (r_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(b_cpu_y, &b_gpu_y);
          }
          r_outdatedOnCpu = false;
          cpu_abswt2S->data[b_bcoef + cpu_abswt2S->size[0] * bcoef] =
              b_cpu_y->data[b_bcoef + b_cpu_y->size[0] * bcoef] /
              fb.Scales->data[b_bcoef];
          r_outdatedOnGpu = true;
        }
      }
    } else {
      gpuEmxMemcpyGpuToCpu_real_T(b_cpu_y, &b_gpu_y);
      binary_expand_op(cpu_abswt2S, b_cpu_y, &fb);
      r_outdatedOnGpu = true;
    }
    i2 = static_cast<int32_T>(defaultSL_idx_1);
    bcoef = c_cpu_y->size[0] * c_cpu_y->size[1];
    c_cpu_y->size[0] = static_cast<int32_T>(defaultSL_idx_1);
    c_cpu_y->size[1] = cpu_abswt2S->size[1];
    emxEnsureCapacity_real_T(c_cpu_y, bcoef, &o_emlrtRTEI);
    j2 = cpu_abswt2S->size[1] - 1;
    mwGetLaunchParameters1D(
        computeNumIters(j2, static_cast<int32_T>(defaultSL_idx_1) - 1), &grid,
        &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_abswt2S, &gpu_abswt2S, !r_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
    if (r_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_abswt2S, cpu_abswt2S);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel9<<<grid, block>>>(
          gpu_abswt2S, static_cast<int32_T>(defaultSL_idx_1) - 1, j2, c_gpu_y,
          c_cpu_y->size[0U], cpu_abswt2S->size[0U]);
    }
    if ((static_cast<int32_T>(defaultSL_idx_1) == 0) &&
        (cpu_abswt2S->size[1] == 0)) {
      if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
        cpsi = fb.Scales->data[0] * 0.0;
      } else {
        cpsi = 0.0;
      }
      bcoef = cpu_savgpTMP->size[0] * cpu_savgpTMP->size[1];
      cpu_savgpTMP->size[0] = 1;
      cpu_savgpTMP->size[1] = 1;
      emxEnsureCapacity_real_T(cpu_savgpTMP, bcoef, &p_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(cpu_savgpTMP, &gpu_savgpTMP, true);
      ec_cwtAvg_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(cpsi,
                                                                  gpu_savgpTMP);
    } else {
      bcoef = cpu_savgpTMP->size[0] * cpu_savgpTMP->size[1];
      cpu_savgpTMP->size[0] = 1;
      cpu_savgpTMP->size[1] = cpu_abswt2S->size[1];
      emxEnsureCapacity_real_T(cpu_savgpTMP, bcoef, &p_emlrtRTEI);
      j2 = cpu_abswt2S->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_savgpTMP, &gpu_savgpTMP, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_cwtAvg_kernel12<<<grid, block>>>(j2, gpu_savgpTMP);
      }
      if (static_cast<int32_T>(defaultSL_idx_1) <= 1) {
        if ((static_cast<int32_T>(defaultSL_idx_1) == 1) &&
            (std::isinf(fb.Scales->data[0]) ||
             std::isnan(fb.Scales->data[0]))) {
          bcoef = cpu_savgpTMP->size[0] * cpu_savgpTMP->size[1];
          cpu_savgpTMP->size[0] = 1;
          cpu_savgpTMP->size[1] = cpu_abswt2S->size[1];
          emxEnsureCapacity_real_T(cpu_savgpTMP, bcoef, &p_emlrtRTEI);
          j2 = cpu_abswt2S->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real_T(cpu_savgpTMP, &gpu_savgpTMP, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_cwtAvg_kernel13<<<grid, block>>>(j2, gpu_savgpTMP);
          }
        }
      } else {
        if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
          r_outdatedOnCpu = false;
          bcoef = cpu_c->size[0];
          cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
          emxEnsureCapacity_real_T(cpu_c, bcoef, &r_emlrtRTEI);
          for (bcoef = 0; bcoef < i2; bcoef++) {
            cpu_c->data[bcoef] = fb.Scales->data[0];
            r_outdatedOnCpu = true;
          }
          cpsi = 0.5 * fb.Scales->data[0];
          gpuEmxEnsureCapacity_real_T(cpu_c, &gpu_c, !r_outdatedOnCpu);
          if (r_outdatedOnCpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_c, cpu_c);
          }
          ec_cwtAvg_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(cpsi,
                                                                      gpu_c);
          r_outdatedOnGpu = false;
          ec_cwtAvg_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              cpsi, static_cast<int32_T>(defaultSL_idx_1), gpu_c);
          r_outdatedOnCpu = false;
        } else {
          bcoef = cpu_c->size[0];
          cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
          emxEnsureCapacity_real_T(cpu_c, bcoef, &q_emlrtRTEI);
          cpu_c->data[0] = 0.5 * (fb.Scales->data[1] - fb.Scales->data[0]);
          for (int32_T k{0}; k <= i2 - 3; k++) {
            cpu_c->data[k + 1] =
                0.5 * (fb.Scales->data[k + 2] - fb.Scales->data[k]);
          }
          cpu_c->data[static_cast<int32_T>(defaultSL_idx_1) - 1] =
              0.5 *
              (fb.Scales->data[static_cast<int32_T>(defaultSL_idx_1) - 1] -
               fb.Scales->data[static_cast<int32_T>(defaultSL_idx_1) - 2]);
          r_outdatedOnCpu = true;
          r_outdatedOnGpu = true;
        }
        if (cpu_abswt2S->size[1] >= 1) {
          cpsi = 0.0;
          alpha1 = 1.0;
          if (r_outdatedOnGpu) {
            gpuEmxEnsureCapacity_real_T(cpu_c, &gpu_c, !r_outdatedOnCpu);
          }
          if (r_outdatedOnCpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_c, cpu_c);
          }
          cublasCheck(hipblasDgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                                  static_cast<int32_T>(defaultSL_idx_1),
                                  cpu_abswt2S->size[1], (double *)&alpha1,
                                  (double *)&c_gpu_y.data[0],
                                  static_cast<int32_T>(defaultSL_idx_1),
                                  (double *)&gpu_c.data[0], 1, (double *)&cpsi,
                                  (double *)&gpu_savgpTMP.data[0], 1),
                      __FILE__, __LINE__);
        }
      }
    }
    bcoef = cpu_xCh->size[0];
    cpu_xCh->size[0] = cpu_savgpTMP->size[1];
    emxEnsureCapacity_real_T(cpu_xCh, bcoef, &s_emlrtRTEI);
    j2 = cpu_savgpTMP->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(j2), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_xCh, &gpu_xCh, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwtAvg_kernel11<<<grid, block>>>(gpu_savgpTMP, j2, gpu_xCh);
    }
    xv_outdatedOnCpu = false;
    xCh_outdatedOnCpu = true;
    //  Downsample
    // 'ec_cwtAvg:38' if doDownsample
    if (doDownsample) {
      // 'ec_cwtAvg:39' y{ch} = resample(xCh,ds(1),ds(2));
      coder::resample(cpu_xCh, &gpu_xCh, &xv_outdatedOnCpu, ds[0], ds[1], cpu_r,
                      &r_outdatedOnCpu, &gpu_r, &r_outdatedOnGpu);
      bcoef = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = cpu_r->size[0];
      emxEnsureCapacity_real_T(y->data[ch].f1, bcoef, &u_emlrtRTEI);
      bcoef = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = 1;
      emxEnsureCapacity_real_T(y->data[ch].f1, bcoef, &u_emlrtRTEI);
      for (bcoef = 0; bcoef < cpu_r->size[0]; bcoef++) {
        if (r_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_r, &gpu_r);
        }
        r_outdatedOnCpu = false;
        y->data[ch].f1->data[bcoef] = cpu_r->data[bcoef];
      }
      //  Downsample
    } else {
      // 'ec_cwtAvg:40' else
      // 'ec_cwtAvg:41' y{ch} = xCh;
      bcoef = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = cpu_xCh->size[0];
      emxEnsureCapacity_real_T(y->data[ch].f1, bcoef, &t_emlrtRTEI);
      bcoef = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = 1;
      emxEnsureCapacity_real_T(y->data[ch].f1, bcoef, &t_emlrtRTEI);
      for (bcoef = 0; bcoef < cpu_xCh->size[0]; bcoef++) {
        if (xCh_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_xCh, &gpu_xCh);
        }
        xCh_outdatedOnCpu = false;
        y->data[ch].f1->data[bcoef] = cpu_xCh->data[bcoef];
      }
    }
  }
  emxFree_real_T(&d_cpu_x);
  emxFree_real_T(&c_cpu_x);
  emxFree_real_T(&cpu_r);
  emxFree_real_T(&cpu_xCh);
  emxFree_real_T(&cpu_c);
  emxFree_real_T(&c_cpu_y);
  emxFree_real_T(&cpu_abswt2S);
  emxFree_real_T(&b_cpu_y);
  emxFree_real_T(&cpu_y);
  emxFree_creal_T(&cpu_cfs);
  emxFree_real_T(&b_cpu_x);
  emxFree_creal_T(&cpu_cfsposdft);
  emxFree_creal_T(&cpu_xposdft);
  emxFree_real_T(&cpu_xv);
  emxFree_real_T(&cpu_savgpTMP);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_x);
  gpuEmxFree_real_T(&gpu_savgpTMP);
  gpuEmxFree_real_T(&gpu_xv);
  gpuEmxFree_creal_T(&gpu_xposdft);
  gpuEmxFree_creal_T(&gpu_cfsposdft);
  gpuEmxFree_real_T(&c_gpu_x);
  gpuEmxFree_creal_T(&gpu_cfs);
  gpuEmxFree_real_T(&gpu_y);
  gpuEmxFree_real_T(&b_gpu_y);
  gpuEmxFree_real_T(&gpu_abswt2S);
  gpuEmxFree_real_T(&c_gpu_y);
  gpuEmxFree_real_T(&gpu_c);
  gpuEmxFree_real_T(&gpu_xCh);
  gpuEmxFree_real_T(&gpu_r);
  gpuEmxFree_real_T(&b_gpu_x);
  gpuEmxFree_real_T(&d_gpu_x);
}

// End of code generation (ec_cwtAvg.cu)

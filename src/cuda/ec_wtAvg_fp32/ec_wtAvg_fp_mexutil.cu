#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wtAvg_fp_mexutil.cu
//
// Code generation for function 'ec_wtAvg_fp_mexutil'
//

// Include files
#include "ec_wtAvg_fp_mexutil.h"
#include "ec_wtAvg_fp_data.h"
#include "ec_wtAvg_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cmath>
#include <cstring>

// Function Definitions
void b_raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                      const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  nvtxRangePushA("#fcn#b_raiseCudaError#" MW_AT_LOCATION);
  len = strlen(file);
  pn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  fn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

void checkCudaError(hipError_t errCode, const char_T *file, uint32_T b_line)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errCode != hipSuccess) {
    nvtxMarkA("#b_raiseCudaError#" MW_AT_LINE);
    b_raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                     hipGetErrorString(errCode));
  }
  nvtxRangePop();
}

uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  nvtxRangePop();
  return numIters;
}

real_T d_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  nvtxRangePushA("#fcn#d_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 0U,
                          (const void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  nvtxRangePop();
  return ret;
}

int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  nvtxRangePushA("#fcn#div_s32#" MW_AT_LOCATION);
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  nvtxRangePop();
  return quotient;
}

real_T emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier *parentId)
{
  real_T y;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#d_emlrt_marshallIn#" MW_AT_LINE);
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  nvtxRangePop();
  return y;
}

real_T emlrt_marshallIn(const mxArray *a__output_of_length_,
                        const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  y = emlrt_marshallIn(emlrtAlias(a__output_of_length_), &thisId);
  emlrtDestroyArray(&a__output_of_length_);
  nvtxRangePop();
  return y;
}

void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                    emxArray_creal32_T *gpu,
                                    boolean_T needsCopy)
{
  creal32_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxEnsureCapacity_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(creal32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(creal32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_real32_T(const emxArray_real32_T *cpu,
                                   emxArray_real32_T *gpu, boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  real32_T *newData;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxEnsureCapacity_real32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_creal32_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxFree_real32_T(emxArray_real32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real32_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                    const emxArray_creal32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyCpuToGpu_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                   const emxArray_real32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyCpuToGpu_real32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                   emxArray_real32_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyGpuToCpu_real32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_creal32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_creal32_T));
  nvtxRangePop();
}

void gpuEmxReset_real32_T(emxArray_real32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real32_T));
  nvtxRangePop();
}

real_T rt_powd_snf(real_T u0, real_T u1)
{
  real_T y;
  nvtxRangePushA("#fcn#rt_powd_snf#" MW_AT_LOCATION);
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else {
    real_T b;
    real_T c;
    b = std::abs(u0);
    c = std::abs(u1);
    if (std::isinf(u1)) {
      if (b == 1.0) {
        y = 1.0;
      } else if (b > 1.0) {
        if (u1 > 0.0) {
          y = rtInf;
        } else {
          y = 0.0;
        }
      } else if (u1 > 0.0) {
        y = 0.0;
      } else {
        y = rtInf;
      }
    } else if (c == 0.0) {
      y = 1.0;
    } else if (c == 1.0) {
      if (u1 > 0.0) {
        y = u0;
      } else {
        y = 1.0 / u0;
      }
    } else if (u1 == 2.0) {
      y = u0 * u0;
    } else if ((u1 == 0.5) && (u0 >= 0.0)) {
      y = std::sqrt(u0);
    } else if ((u0 < 0.0) && (u1 > std::floor(u1))) {
      y = rtNaN;
    } else {
      y = std::pow(u0, u1);
    }
  }
  nvtxRangePop();
  return y;
}

// End of code generation (ec_wtAvg_fp_mexutil.cu)

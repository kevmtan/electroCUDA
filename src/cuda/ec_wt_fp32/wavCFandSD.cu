#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wavCFandSD.cu
//
// Code generation for function 'wavCFandSD'
//

// Include files
#include "wavCFandSD.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "gammaln.h"
#include "quadgk.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo oc_emlrtRTEI{
    64,                // lineNo
    28,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    64,                // lineNo
    31,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    1,            // lineNo
    39,           // colNo
    "wavCFandSD", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavCFandSD.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    64,                // lineNo
    12,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
real_T wavCFandSD(char_T wname[5], real_T varargin_1, real_T varargin_2,
                  real_T *sigmaT, real_T *cf)
{
  emxArray_real_T *b_dv1;
  emxArray_real_T *b_x;
  emxArray_real_T *b_y;
  emxArray_real_T *fx;
  emxArray_real_T *x;
  emxArray_real_T *xt;
  emxArray_real_T *y;
  real_T interval[650];
  real_T FourierFactor;
  real_T b_be;
  real_T d;
  real_T d1;
  real_T d2;
  real_T d3;
  real_T d4;
  real_T err_ok;
  int32_T k;
  nvtxRangePushA("#fcn#wavCFandSD#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  profileLoopStart("wavCFandSD_loop_0", __LINE__, 4 + 1, "");
  for (k = 0; k < 5; k++) {
    wname[k] = cv[static_cast<uint8_T>(wname[k]) & 127];
  }
  profileLoopEnd();
  if (wname[0] == 'm') {
    real_T abserrsubk;
    real_T be;
    real_T halfh;
    real_T intFsq;
    real_T midpt;
    real_T q_ok;
    real_T tol;
    *cf = std::exp(1.0 / varargin_1 *
                   (std::log(varargin_2) - std::log(varargin_1)));
    midpt = 2.0 * (varargin_2 - 1.0);
    halfh = 2.0 * varargin_2;
    abserrsubk = (varargin_2 - 1.0) + varargin_1;
    tol = 2.0 * ((varargin_2 - 1.0) + varargin_1);
    intFsq = 2.0 * varargin_2;
    be = (varargin_2 - 1.0) + varargin_1 / 2.0;
    b_be = 2.0 * ((varargin_2 - 1.0) + varargin_1 / 2.0);
    q_ok = 2.0 * varargin_2;
    d = (2.0 * (varargin_2 - 1.0) + 1.0) / varargin_1;
    nvtxMarkA("#gammaln#" MW_AT_LINE);
    gammaln(&d);
    d1 = (2.0 * varargin_2 + 1.0) / varargin_1;
    nvtxMarkA("#gammaln#" MW_AT_LINE);
    gammaln(&d1);
    err_ok = (2.0 * ((varargin_2 - 1.0) + varargin_1) + 1.0) / varargin_1;
    nvtxMarkA("#gammaln#" MW_AT_LINE);
    gammaln(&err_ok);
    d2 = (2.0 * varargin_2 + 1.0) / varargin_1;
    nvtxMarkA("#gammaln#" MW_AT_LINE);
    gammaln(&d2);
    d3 = (2.0 * ((varargin_2 - 1.0) + varargin_1 / 2.0) + 1.0) / varargin_1;
    nvtxMarkA("#gammaln#" MW_AT_LINE);
    gammaln(&d3);
    d4 = (2.0 * varargin_2 + 1.0) / varargin_1;
    nvtxMarkA("#gammaln#" MW_AT_LINE);
    gammaln(&d4);
    *sigmaT = std::sqrt(
        (std::exp(((((((2.0 * (varargin_2 / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(varargin_2))) -
                        2.0 * ((varargin_2 - 1.0) / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(varargin_2 - 1.0)))) +
                       midpt / varargin_1 *
                           ((std::log(varargin_1) + 1.0) - std::log(midpt))) -
                      halfh / varargin_1 *
                          ((std::log(varargin_1) + 1.0) - std::log(halfh))) +
                     2.0 / varargin_1 * std::log(varargin_2 / varargin_1)) +
                    2.0 * std::log(varargin_2)) +
                   d) -
                  d1) +
         std::exp(((((((2.0 * (varargin_2 / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(varargin_2))) -
                        2.0 * (abserrsubk / varargin_1 *
                               ((std::log(varargin_1) + 1.0) -
                                std::log(abserrsubk)))) +
                       tol / varargin_1 *
                           ((std::log(varargin_1) + 1.0) - std::log(tol))) -
                      intFsq / varargin_1 *
                          ((std::log(varargin_1) + 1.0) - std::log(intFsq))) +
                     2.0 / varargin_1 * std::log(varargin_2 / varargin_1)) +
                    2.0 * std::log(varargin_1)) +
                   err_ok) -
                  d2)) -
        std::exp(
            ((((((((2.0 * (varargin_2 / varargin_1 *
                           ((std::log(varargin_1) + 1.0) -
                            std::log(varargin_2))) -
                    2.0 * (be / varargin_1 *
                           ((std::log(varargin_1) + 1.0) - std::log(be)))) +
                   b_be / varargin_1 *
                       ((std::log(varargin_1) + 1.0) - std::log(b_be))) -
                  q_ok / varargin_1 *
                      ((std::log(varargin_1) + 1.0) - std::log(q_ok))) +
                 2.0 / varargin_1 * std::log(varargin_2 / varargin_1)) +
                0.69314718055994529) +
               std::log(varargin_2)) +
              std::log(varargin_1)) +
             d3) -
            d4));
    if (std::isinf(*sigmaT) || std::isnan(*sigmaT)) {
      real_T subs[1298];
      real_T errsub[649];
      real_T qsub[649];
      int32_T ix;
      be = std::exp(1.0 / varargin_1 *
                    (std::log(varargin_2) - std::log(varargin_1)));
      interval[0] = 0.0;
      interval[1] = 1.0;
      std::memset(&interval[2], 0, 648U * sizeof(real_T));
      intFsq = 0.0;
      nvtxMarkA("#split#" MW_AT_LINE);
      ix = split(interval, 2, &b_be);
      if (!(b_be > 0.0)) {
        nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
        nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
        intFsq = rtInf * (rt_powd_snf(rtInf, 2.0 * varargin_2) *
                          std::exp(-2.0 * rt_powd_snf(rtInf, varargin_1)));
      } else {
        int32_T nsubs;
        boolean_T first_iteration;
        nsubs = ix - 2;
        profileLoopStart("wavCFandSD_loop_1", __LINE__, nsubs + 1, "");
        for (k = 0; k <= nsubs; k++) {
          subs[k << 1] = interval[k];
          subs[(k << 1) + 1] = interval[k + 1];
        }
        profileLoopEnd();
        q_ok = 0.0;
        err_ok = 0.0;
        first_iteration = true;
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&x, 2, &lc_emlrtRTEI, true);
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&b_x, 2, &rc_emlrtRTEI, true);
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&xt, 2, &sc_emlrtRTEI, true);
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&y, 2, &tc_emlrtRTEI, true);
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&fx, 2, &uc_emlrtRTEI, true);
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&b_dv1, 2, &vc_emlrtRTEI, true);
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&b_y, 2, &vc_emlrtRTEI, true);
        nvtxRangePushA("#loop#wavCFandSD_whileloop_0##" MW_AT_LINE);
        int32_T exitg1;
        do {
          boolean_T guard1;
          exitg1 = 0;
          ix = x->size[0] * x->size[1];
          x->size[0] = 1;
          x->size[1] = 15 * (nsubs + 1);
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(x, ix, &lc_emlrtRTEI);
          ix = -1;
          profileLoopStart("wavCFandSD_loop_2", __LINE__, nsubs + 1, "");
          for (k = 0; k <= nsubs; k++) {
            d = subs[k << 1];
            d1 = subs[(k << 1) + 1];
            midpt = (d + d1) / 2.0;
            halfh = (d1 - d) / 2.0;
            profileLoopStart("wavCFandSD_loop_3", __LINE__, 14 + 1, "");
            for (int32_T j{0}; j < 15; j++) {
              x->data[(ix + j) + 1] = dv[j] * halfh + midpt;
            }
            profileLoopEnd();
            ix += 15;
          }
          profileLoopEnd();
          ix = b_x->size[0] * b_x->size[1];
          b_x->size[0] = 1;
          b_x->size[1] = x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_x, ix, &mc_emlrtRTEI);
          ix = xt->size[0] * xt->size[1];
          xt->size[0] = 1;
          xt->size[1] = x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(xt, ix, &nc_emlrtRTEI);
          ix = x->size[1];
          profileLoopStart("wavCFandSD_loop_4", __LINE__, (ix - 1) + 1, "");
          for (k = 0; k < ix; k++) {
            d = x->data[k];
            midpt = d / (1.0 - d);
            b_x->data[k] = midpt * midpt;
            xt->data[k] = 2.0 * midpt / ((1.0 - d) * (1.0 - d));
          }
          profileLoopEnd();
          guard1 = false;
          if (!first_iteration) {
            boolean_T exitg2;
            midpt = std::abs(b_x->data[0]);
            k = 0;
            exitg2 = false;
            nvtxRangePushA("#loop#wavCFandSD_whileloop_1##" MW_AT_LINE);
            while ((!exitg2) && (k <= b_x->size[1] - 2)) {
              halfh = midpt;
              midpt = std::abs(b_x->data[k + 1]);
              if (std::abs(b_x->data[k + 1] - b_x->data[k]) <=
                  2.2204460492503131E-14 * std::fmax(halfh, midpt)) {
                first_iteration = true;
                exitg2 = true;
              } else {
                k++;
              }
            }
            nvtxRangePop();
            if (first_iteration) {
              int16_T dv_idx_1;
              dv_idx_1 = static_cast<int16_T>(x->size[1]);
              ix = fx->size[0] * fx->size[1];
              fx->size[0] = 1;
              fx->size[1] = x->size[1];
              nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
              emxEnsureCapacity_real_T(fx, ix, &pc_emlrtRTEI);
              profileLoopStart("wavCFandSD_loop_8", __LINE__,
                               (dv_idx_1 - 1) + 1, "");
              for (ix = 0; ix < dv_idx_1; ix++) {
                fx->data[ix] = 0.0;
              }
              profileLoopEnd();
            } else {
              guard1 = true;
            }
          } else {
            guard1 = true;
          }
          if (guard1) {
            first_iteration = false;
            ix = y->size[0] * y->size[1];
            y->size[0] = 1;
            y->size[1] = b_x->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(y, ix, &m_emlrtRTEI);
            ix = b_x->size[1];
            profileLoopStart("wavCFandSD_loop_6", __LINE__, (ix - 1) + 1, "");
            for (k = 0; k < ix; k++) {
              nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
              y->data[k] = rt_powd_snf(b_x->data[k], varargin_1);
            }
            profileLoopEnd();
            ix = b_dv1->size[0] * b_dv1->size[1];
            b_dv1->size[0] = 1;
            b_dv1->size[1] = y->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(b_dv1, ix, &oc_emlrtRTEI);
            profileLoopStart("wavCFandSD_loop_10", __LINE__,
                             (y->size[1] - 1) + 1, "");
            for (ix = 0; ix < y->size[1]; ix++) {
              b_dv1->data[ix] = -2.0 * y->data[ix];
            }
            profileLoopEnd();
            ix = b_x->size[1];
            profileLoopStart("wavCFandSD_loop_11", __LINE__, (ix - 1) + 1, "");
            for (k = 0; k < ix; k++) {
              b_dv1->data[k] = std::exp(b_dv1->data[k]);
            }
            profileLoopEnd();
            midpt = 2.0 * varargin_2;
            ix = b_y->size[0] * b_y->size[1];
            b_y->size[0] = 1;
            b_y->size[1] = b_x->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(b_y, ix, &m_emlrtRTEI);
            ix = b_x->size[1];
            profileLoopStart("wavCFandSD_loop_13", __LINE__, (ix - 1) + 1, "");
            for (k = 0; k < ix; k++) {
              nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
              b_y->data[k] = rt_powd_snf(b_x->data[k], midpt);
            }
            profileLoopEnd();
            if (b_y->size[1] == b_dv1->size[1]) {
              ix = fx->size[0] * fx->size[1];
              fx->size[0] = 1;
              fx->size[1] = b_y->size[1];
              nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
              emxEnsureCapacity_real_T(fx, ix, &qc_emlrtRTEI);
              profileLoopStart("wavCFandSD_loop_14", __LINE__,
                               (b_y->size[1] - 1) + 1, "");
              for (ix = 0; ix < b_y->size[1]; ix++) {
                fx->data[ix] = b_y->data[ix] * b_dv1->data[ix] * xt->data[ix];
              }
              profileLoopEnd();
            } else {
              nvtxMarkA("#binary_expand_op_3#" MW_AT_LINE);
              binary_expand_op_3(fx, b_y, b_dv1, xt);
            }
          }
          if (first_iteration) {
            exitg1 = 1;
          } else {
            midpt = 0.0;
            ix = -1;
            profileLoopStart("wavCFandSD_loop_5", __LINE__, nsubs + 1, "");
            for (k = 0; k <= nsubs; k++) {
              d = 0.0;
              d1 = 0.0;
              profileLoopStart("wavCFandSD_loop_7", __LINE__, 14 + 1, "");
              for (int32_T j{0}; j < 15; j++) {
                d += dv1[j] * fx->data[(ix + j) + 1];
                d1 += dv2[j] * fx->data[(ix + j) + 1];
              }
              profileLoopEnd();
              ix += 15;
              halfh = (subs[(k << 1) + 1] - subs[k << 1]) / 2.0;
              d *= halfh;
              qsub[k] = d;
              midpt += d;
              errsub[k] = d1 * halfh;
            }
            profileLoopEnd();
            intFsq = midpt + q_ok;
            tol = std::fmax(1.0E-10, 1.0E-6 * std::abs(intFsq));
            midpt = 2.0 * tol / b_be;
            halfh = 0.0;
            ix = 0;
            profileLoopStart("wavCFandSD_loop_9", __LINE__, nsubs + 1, "");
            for (k = 0; k <= nsubs; k++) {
              d = errsub[k];
              abserrsubk = std::abs(d);
              if (abserrsubk <=
                  midpt * ((subs[(k << 1) + 1] - subs[k << 1]) / 2.0)) {
                err_ok += d;
                q_ok += qsub[k];
              } else {
                halfh += abserrsubk;
                ix++;
                subs[(ix - 1) << 1] = subs[k << 1];
                subs[((ix - 1) << 1) + 1] = subs[(k << 1) + 1];
              }
            }
            profileLoopEnd();
            midpt = std::abs(err_ok) + halfh;
            if ((!std::isinf(intFsq)) && (!std::isnan(intFsq)) &&
                ((!std::isinf(midpt)) && (!std::isnan(midpt))) && (ix != 0) &&
                (!(midpt <= tol))) {
              nsubs = (ix << 1) - 1;
              if (nsubs + 1 > 650) {
                exitg1 = 1;
              } else {
                profileLoopStart("wavCFandSD_loop_12", __LINE__, (ix - 1) + 1,
                                 "");
                for (k = 0; k < ix; k++) {
                  subs[((((ix - k) << 1) - 1) << 1) + 1] =
                      subs[(((ix - k) - 1) << 1) + 1];
                  subs[(((ix - k) << 1) - 1) << 1] =
                      (subs[((ix - k) - 1) << 1] +
                       subs[(((ix - k) - 1) << 1) + 1]) /
                      2.0;
                  subs[((((ix - k) << 1) - 2) << 1) + 1] =
                      subs[(((ix - k) << 1) - 1) << 1];
                  subs[(((ix - k) << 1) - 2) << 1] = subs[((ix - k) - 1) << 1];
                }
                profileLoopEnd();
              }
            } else {
              exitg1 = 1;
            }
          }
        } while (exitg1 == 0);
        nvtxRangePop();
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&b_y);
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&b_dv1);
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&fx);
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&y);
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&xt);
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&b_x);
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&x);
      }
      nvtxMarkA("#quadgk#" MW_AT_LINE);
      *sigmaT = std::sqrt(be * be * (quadgk(varargin_2, varargin_1) / intFsq));
    }
  } else if (wname[0] == 'a') {
    *cf = 6.0;
    *sigmaT = 1.4142135623730951;
  } else {
    *cf = 5.0;
    *sigmaT = 5.847705;
  }
  FourierFactor = 6.2831853071795862 / *cf;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
  return FourierFactor;
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (wavCFandSD.cu)

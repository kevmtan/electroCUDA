#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// conv2.cu
//
// Code generation for function 'conv2'
//

// Include files
#include "conv2.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo xg_emlrtRTEI{
    49,                                                               // lineNo
    31,                                                               // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo yg_emlrtRTEI{
    47,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo ah_emlrtRTEI{
    58,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo bh_emlrtRTEI{
    75,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo
    ch_emlrtRTEI{
        88,                  // lineNo
        13,                  // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo dh_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "stencil_codegen", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "stencil_codegen.p" // pName
};

static emlrtRTEInfo eh_emlrtRTEI{
    49,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo fh_emlrtRTEI{
    172,                                                          // lineNo
    20,                                                           // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

static emlrtRTEInfo gh_emlrtRTEI{
    49,                                                               // lineNo
    13,                                                               // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo hh_emlrtRTEI{
    82,                                                           // lineNo
    9,                                                            // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

static emlrtRTEInfo ih_emlrtRTEI{
    56,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

// Function Declarations
namespace coder {
static void
conv2NonSeparable(emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
                  emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
                  emxArray_creal32_T *cpu_b, emxArray_creal32_T *gpu_b,
                  boolean_T *b_outdatedOnGpu, emxArray_creal32_T *cpu_c,
                  boolean_T *c_outdatedOnCpu, emxArray_creal32_T *gpu_c,
                  boolean_T *c_outdatedOnGpu);

}
static __global__ void conv2NonSeparable_kernel73(const int32_T b,
                                                  emxArray_creal32_T c);

static __global__ void conv2NonSeparable_kernel74(const int32_T b,
                                                  emxArray_creal32_T c);

static __global__ void conv2NonSeparable_kernel75(const real32_T f,
                                                  creal32_T a_data[1]);

static __global__ void conv2NonSeparable_kernel76(const emxArray_creal32_T b,
                                                  const creal32_T a_data[1],
                                                  emxArray_creal32_T c);

static __global__ void conv2NonSeparable_kernel77(const int32_T OH,
                                                  emxArray_real32_T expanded);

static __global__ void conv2NonSeparable_kernel78(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void
conv2NonSeparable_kernel79(const emxArray_real32_T a, const int32_T mc,
                           const emxArray_int32_T iv, const int32_T b,
                           emxArray_real32_T expanded, int32_T expanded_dim0);

static __global__ void conv2NonSeparable_kernel80(
    const emxArray_real32_T expanded, const emxArray_int32_T cols,
    const emxArray_int32_T rows, const emxArray_creal32_T b, const int32_T c,
    const int32_T d, emxArray_creal32_T b_c, int32_T b_dim0, int32_T b_dim1,
    int32_T expanded_dim0, int32_T c_dim0);

static __global__ void conv2_kernel52(const emxArray_creal32_T a,
                                      const int32_T b_a, emxArray_creal32_T c_a,
                                      int32_T a_dim0);

static __global__ void conv2_kernel53(const int32_T a, emxArray_creal32_T c);

static __global__ void conv2_kernel54(const emxArray_creal32_T a,
                                      const real32_T cv_im, real32_T *f,
                                      real32_T *cv_re);

static __global__ void conv2_kernel55(const real32_T *cv_re, const real32_T *f,
                                      emxArray_creal32_T c);

static __global__ void conv2_kernel56(const emxArray_creal32_T c,
                                      const int32_T b_c, emxArray_creal32_T c_c,
                                      int32_T c_dim0);

static __global__ void conv2_kernel57(const int32_T a,
                                      emxArray_creal32_T expanded);

static __global__ void conv2_kernel58(const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel59(const emxArray_creal32_T a,
                                      const emxArray_int32_T iv,
                                      const int32_T b,
                                      emxArray_creal32_T expanded);

static __global__ void conv2_kernel60(const emxArray_creal32_T expanded,
                                      const real32_T cv_im, const int32_T b,
                                      emxArray_creal32_T c);

static __global__ void conv2_kernel61(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel62(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel63(const real32_T cv_re,
                                      creal32_T b_data[1]);

static __global__ void conv2_kernel64(const emxArray_creal32_T a,
                                      const creal32_T b_data[1],
                                      emxArray_creal32_T b_a);

static __global__ void conv2_kernel65(const emxArray_creal32_T a,
                                      const int32_T iv1, emxArray_creal32_T c);

static __global__ void conv2_kernel66(const int32_T OH,
                                      emxArray_creal32_T expanded);

static __global__ void conv2_kernel67(const int32_T offsetH,
                                      const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv1);

static __global__ void conv2_kernel68(const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv2);

static __global__ void
conv2_kernel69(const emxArray_creal32_T a, const int32_T iv1_dim0,
               const emxArray_int32_T iv2, const emxArray_int32_T iv1,
               const int32_T b_iv1, const int32_T c_iv1,
               emxArray_creal32_T expanded, int32_T expanded_dim0);

static __global__ void
conv2_kernel70(const int32_T offsetH, const emxArray_creal32_T expanded,
               const int32_T k, const emxArray_int32_T rows,
               const int32_T b_rows, emxArray_creal32_T newIm,
               int32_T expanded_dim0);

static __global__ void conv2_kernel71(const real32_T cv_re,
                                      const int32_T offsetH, const int32_T k,
                                      emxArray_creal32_T c, int32_T c_dim0);

static __global__ void conv2_kernel72(const real32_T cv_im,
                                      const int32_T offsetH, const int32_T k,
                                      emxArray_creal32_T c, int32_T c_dim0);

static __global__ void conv2_kernel81(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel82(const int32_T OH,
                                      emxArray_real32_T expanded);

static __global__ void conv2_kernel83(const int32_T offsetH,
                                      const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv1);

static __global__ void conv2_kernel84(const emxArray_real32_T b,
                                      const emxArray_int32_T iv1,
                                      const int32_T c,
                                      emxArray_real32_T expanded);

static __global__ void conv2_kernel85(const emxArray_real32_T expanded,
                                      const emxArray_int32_T rows,
                                      const emxArray_creal32_T a,
                                      const int32_T b, emxArray_creal32_T c,
                                      int32_T a_dim0);

static __global__ void conv2_kernel86(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel87(const int32_T OH,
                                      emxArray_creal32_T expanded);

static __global__ void conv2_kernel88(const int32_T offsetH,
                                      const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel89(const emxArray_creal32_T a,
                                      const emxArray_int32_T iv,
                                      const int32_T b,
                                      emxArray_creal32_T expanded);

static __global__ void conv2_kernel90(const emxArray_creal32_T expanded,
                                      const int32_T offsetH,
                                      const emxArray_int32_T rows,
                                      const int32_T b_rows,
                                      emxArray_creal32_T newIm);

static __global__ void conv2_kernel91(const real32_T cv_im,
                                      const real32_T cv_re,
                                      const int32_T offsetH,
                                      emxArray_creal32_T c);

// Function Definitions
//
//
namespace coder {
static void
conv2NonSeparable(emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
                  emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
                  emxArray_creal32_T *cpu_b, emxArray_creal32_T *gpu_b,
                  boolean_T *b_outdatedOnGpu, emxArray_creal32_T *cpu_c,
                  boolean_T *c_outdatedOnCpu, emxArray_creal32_T *gpu_c,
                  boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_cols;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *cpu_cols;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *cpu_expanded;
  creal32_T(*gpu_a_data)[1];
  int32_T k;
  int32_T mc;
  int32_T nc;
  boolean_T validLaunchParams;
  nvtxRangePushA("#fcn#conv2NonSeparable#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_a_data, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_cols);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_expanded);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  if (cpu_b->size[1] == 0) {
    nc = 1;
  } else {
    nc = cpu_b->size[1];
  }
  k = cpu_c->size[0] * cpu_c->size[1];
  cpu_c->size[0] = mc;
  cpu_c->size[1] = nc;
  nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
  emxEnsureCapacity_creal32_T(cpu_c, k, &bh_emlrtRTEI);
  k = mc * nc - 1;
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#conv2NonSeparable_kernel73#" MW_AT_LINE);
    conv2NonSeparable_kernel73<<<grid, block>>>(k, *gpu_c);
  }
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_expanded, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &hh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_cols, 2, &dh_emlrtRTEI, true);
  if ((cpu_a->size[0] == 0) ||
      ((cpu_b->size[0] == 0) || (cpu_b->size[1] == 0))) {
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_c, k, &bh_emlrtRTEI);
    k = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel74#" MW_AT_LINE);
      conv2NonSeparable_kernel74<<<grid, block>>>(k, *gpu_c);
    }
  } else if ((cpu_a->size[0] == 1) &&
             ((cpu_b->size[0] == 1) && (cpu_b->size[1] == 1))) {
    real32_T f;
    if (*a_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
    }
    *a_outdatedOnCpu = false;
    f = cpu_a->data[0];
    nvtxMarkA("#conv2NonSeparable_kernel75#" MW_AT_LINE);
    conv2NonSeparable_kernel75<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        f, *gpu_a_data);
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = 1;
    cpu_c->size[1] = 1;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_c, k, &bh_emlrtRTEI);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    nvtxMarkA("#conv2NonSeparable_kernel76#" MW_AT_LINE);
    conv2NonSeparable_kernel76<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_b, *gpu_a_data, *gpu_c);
  } else {
    real_T blockDims[3];
    int32_T OW;
    int32_T b_OH;
    int32_T offsetH;
    uint32_T OH;
    int8_T threadDims[3];
    OH = (static_cast<uint32_T>(cpu_a->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    nc = static_cast<int32_T>(
        std::fmin(32.0, (static_cast<real_T>(cpu_b->size[1]) + 1.0) - 1.0));
    blockDims[0] =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
                   static_cast<real_T>(mc));
    blockDims[1] =
        std::floor((static_cast<real_T>(static_cast<uint32_T>(cpu_b->size[1]) +
                                        static_cast<uint32_T>(nc)) -
                    1.0) /
                   static_cast<real_T>(nc));
    threadDims[0] = static_cast<int8_T>(mc);
    threadDims[1] = static_cast<int8_T>(nc);
    OW = cpu_b->size[1];
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    b_OH = static_cast<int32_T>(OH);
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
      mc = MAX_int32_T;
    } else {
      mc = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[0] = mc - 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_expanded, k, &dh_emlrtRTEI);
    if (cpu_b->size[1] > MAX_int32_T - cpu_b->size[1]) {
      mc = MAX_int32_T;
    } else {
      mc = cpu_b->size[1] + cpu_b->size[1];
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[1] = mc - 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_expanded, k, &dh_emlrtRTEI);
    k = ((static_cast<int32_T>(OH) + cpu_b->size[0]) - 1) *
            ((cpu_b->size[1] + cpu_b->size[1]) - 1) -
        1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel77#" MW_AT_LINE);
      conv2NonSeparable_kernel77<<<grid, block>>>(k, gpu_expanded);
    }
    nc = cpu_a->size[0];
    k = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_a->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_y, k, &fh_emlrtRTEI);
    cpu_y->data[0] = 1;
    mc = 1;
    profileLoopStart("conv2NonSeparable_loop_0", __LINE__, (nc - 2) + 1, "");
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_y->data[k + 1] = mc;
    }
    profileLoopEnd();
    k = cpu_iv->size[0];
    cpu_iv->size[0] = cpu_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_iv, k, &dh_emlrtRTEI);
    k = cpu_y->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel78#" MW_AT_LINE);
      conv2NonSeparable_kernel78<<<grid, block>>>(offsetH, gpu_y, k, gpu_iv);
    }
    mc = static_cast<int32_T>(
             std::floor(static_cast<real_T>(cpu_b->size[1]) / 2.0)) +
         static_cast<int32_T>(
             std::floor((static_cast<real_T>(cpu_b->size[1]) - 1.0) / 2.0));
    nc = cpu_iv->size[0];
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    if (*a_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel79#" MW_AT_LINE);
      conv2NonSeparable_kernel79<<<grid, block>>>(
          *gpu_a, mc, gpu_iv, nc - 1, gpu_expanded, cpu_expanded->size[0U]);
    }
    nc = cpu_b->size[0];
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = cpu_b->size[0];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_rows, k, &fh_emlrtRTEI);
    cpu_rows->data[0] = 0;
    mc = 0;
    profileLoopStart("conv2NonSeparable_loop_1", __LINE__, (nc - 2) + 1, "");
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_rows->data[k + 1] = mc;
    }
    profileLoopEnd();
    nc = cpu_b->size[1];
    k = cpu_cols->size[0] * cpu_cols->size[1];
    cpu_cols->size[0] = 1;
    cpu_cols->size[1] = cpu_b->size[1];
    nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
    emxEnsureCapacity_int32_T(cpu_cols, k, &fh_emlrtRTEI);
    cpu_cols->data[0] = 0;
    mc = 0;
    profileLoopStart("conv2NonSeparable_loop_2", __LINE__, (nc - 2) + 1, "");
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_cols->data[k + 1] = mc;
    }
    profileLoopEnd();
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = static_cast<int32_T>(OH);
    cpu_c->size[1] = cpu_b->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_c, k, &dh_emlrtRTEI);
    if (blockDims[0] < 4.294967296E+9) {
      OH = static_cast<uint32_T>(blockDims[0]);
    } else {
      OH = MAX_uint32_T;
    }
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                            dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                            dim3(static_cast<uint32_T>(threadDims[0]),
                                 static_cast<uint32_T>(threadDims[1]), 1U),
                            &grid, &block);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_cols, &gpu_cols, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_cols, cpu_cols);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2NonSeparable_kernel80#" MW_AT_LINE);
      conv2NonSeparable_kernel80<<<grid, block>>>(
          gpu_expanded, gpu_cols, gpu_rows, *gpu_b, b_OH - 1, OW - 1, *gpu_c,
          cpu_b->size[0U], cpu_b->size[1U], cpu_expanded->size[0U],
          cpu_c->size[0U]);
    }
  }
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_cols);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_cols);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_a_data), __FILE__, __LINE__);
  *c_outdatedOnCpu = true;
  *c_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace coder
static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel73(
    const int32_T b, emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0F;
    c.data[qY].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel74(
    const int32_T b, emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0F;
    c.data[qY].im = 0.0F;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel75(
    const real32_T f, creal32_T a_data[1])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    a_data[0].im = 0.0F;
    a_data[0].re = f;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel76(
    const emxArray_creal32_T b, const creal32_T a_data[1], emxArray_creal32_T c)
{
  int32_T k;
  k = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (k < 1) {
    creal32_T b_b;
    b_b.re = a_data[0].re * b.data[0].re - 0.0F * b.data[0].im;
    b_b.im = a_data[0].re * b.data[0].im + 0.0F * b.data[0].re;
    c.data[0] = b_b;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel77(
    const int32_T OH, emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[qY] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel78(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T q1;
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    q1 = y.data[qY];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[qY] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel79(
    const emxArray_real32_T a, const int32_T mc, const emxArray_int32_T iv,
    const int32_T b, emxArray_real32_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[iv.data[qY] + expanded_dim0 * mc] = a.data[qY];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel80(
    const emxArray_real32_T expanded, const emxArray_int32_T cols,
    const emxArray_int32_T rows, const emxArray_creal32_T b, const int32_T c,
    const int32_T d, emxArray_creal32_T b_c, int32_T b_dim0, int32_T b_dim1,
    int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(d) + 1UL) * (static_cast<uint64_T>(c) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    creal32_T b_cv;
    int32_T ocol;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(c) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(c) + 1UL));
    b_cv.re = 0.0F;
    b_cv.im = 0.0F;
    for (int32_T n{0}; n < b_dim1; n++) {
      for (int32_T m{0}; m < b_dim0; m++) {
        creal32_T c_b;
        int32_T q1;
        int32_T qY;
        c_b = b.data[((b_dim0 - m) + b_dim0 * ((b_dim1 - n) - 1)) - 1];
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv.re += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] * c_b.re;
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv.im += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] * c_b.im;
      }
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel52(
    const emxArray_creal32_T a, const int32_T b_a, emxArray_creal32_T c_a,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_a.data[i1] = a.data[a_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel53(const int32_T a,
                                                           emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1].re = 0.0F;
    c.data[i1].im = 0.0F;
  }
}

static __global__
    __launch_bounds__(32, 1) void conv2_kernel54(const emxArray_creal32_T a,
                                                 const real32_T cv_im,
                                                 real32_T *f, real32_T *cv_re)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *cv_re = cv_im * a.data[0].re;
    *f = cv_im * a.data[0].im;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel55(
    const real32_T *cv_re, const real32_T *f, emxArray_creal32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0].im = *f;
    c.data[0].re = *cv_re;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel56(
    const emxArray_creal32_T c, const int32_T b_c, emxArray_creal32_T c_c,
    int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_c.data[c_dim0 * i1] = c.data[i1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel57(const int32_T a,
                                                   emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    expanded.data[i1].re = 0.0F;
    expanded.data[i1].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel58(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    iv.data[i1] = y.data[i1] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel59(
    const emxArray_creal32_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    expanded.data[iv.data[i1]] = a.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel60(
    const emxArray_creal32_T expanded, const real32_T cv_im, const int32_T b,
    emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T q1;
    q1 = static_cast<int32_T>(idx);
    c.data[q1].re = cv_im * expanded.data[q1].re;
    c.data[q1].im = cv_im * expanded.data[q1].im;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel61(const int32_T b,
                                                           emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1].re = 0.0F;
    c.data[i1].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel62(const int32_T b,
                                                           emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1].re = 0.0F;
    c.data[i1].im = 0.0F;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void conv2_kernel63(const real32_T cv_re,
                                                           creal32_T b_data[1])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0].im = 0.0F;
    b_data[0].re = cv_re;
  }
}

static __global__
    __launch_bounds__(32, 1) void conv2_kernel64(const emxArray_creal32_T a,
                                                 const creal32_T b_data[1],
                                                 emxArray_creal32_T b_a)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 1) {
    real32_T f1;
    b_a.data[0].re = 0.0F;
    b_a.data[0].im = 0.0F;
    f1 = b_data[0].re;
    b_a.data[0].re += a.data[0].re * f1 - a.data[0].im * 0.0F;
    b_a.data[0].im += a.data[0].re * 0.0F + a.data[0].im * f1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel65(
    const emxArray_creal32_T a, const int32_T iv1, emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(iv1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1] = a.data[i1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel66(const int32_T OH,
                                                   emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    expanded.data[i1].re = 0.0F;
    expanded.data[i1].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel67(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    i1 = static_cast<int32_T>(idx);
    q1 = y.data[i1];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[i1] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel68(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv2)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    iv2.data[i1] = y.data[i1] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel69(
    const emxArray_creal32_T a, const int32_T iv1_dim0,
    const emxArray_int32_T iv2, const emxArray_int32_T iv1, const int32_T b_iv1,
    const int32_T c_iv1, emxArray_creal32_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_iv1) + 1UL) *
                (static_cast<uint64_T>(b_iv1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    q1 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_iv1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(q1)) /
                              (static_cast<uint64_T>(b_iv1) + 1UL));
    expanded.data[iv1.data[q1] + expanded_dim0 * iv2.data[i1]] =
        a.data[q1 + iv1_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel70(
    const int32_T offsetH, const emxArray_creal32_T expanded, const int32_T k,
    const emxArray_int32_T rows, const int32_T b_rows, emxArray_creal32_T newIm,
    int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_rows);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    i1 = static_cast<int32_T>(idx);
    q1 = rows.data[i1];
    if (q1 > 2147483646 - k) {
      q1 = MAX_int32_T;
    } else {
      q1 = (k + q1) + 1;
    }
    newIm.data[i1] = expanded.data[(q1 + expanded_dim0 * offsetH) - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel71(
    const real32_T cv_re, const int32_T offsetH, const int32_T k,
    emxArray_creal32_T c, int32_T c_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[k + c_dim0 * offsetH].re = cv_re;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel72(
    const real32_T cv_im, const int32_T offsetH, const int32_T k,
    emxArray_creal32_T c, int32_T c_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[k + c_dim0 * offsetH].im = cv_im;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel81(const int32_T b,
                                                           emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow].re = 0.0F;
    c.data[orow].im = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel82(const int32_T OH,
                                                   emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel83(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel84(
    const emxArray_real32_T b, const emxArray_int32_T iv1, const int32_T c,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv1.data[orow]] = b.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel85(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_creal32_T a, const int32_T b, emxArray_creal32_T c,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    creal32_T b_cv;
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_cv.re = 0.0F;
    b_cv.im = 0.0F;
    for (int32_T m{0}; m < a_dim0; m++) {
      creal32_T b_b;
      int32_T q1;
      b_b = a.data[(a_dim0 - m) - 1];
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv.re += expanded.data[q1 - 1] * b_b.re;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv.im += expanded.data[q1 - 1] * b_b.im;
    }
    c.data[orow] = b_cv;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel86(const int32_T b,
                                                           emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow].re = 0.0F;
    c.data[orow].im = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel87(const int32_T OH,
                                                   emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow].re = 0.0F;
    expanded.data[orow].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel88(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel89(
    const emxArray_creal32_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv.data[orow]] = a.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel90(
    const emxArray_creal32_T expanded, const int32_T offsetH,
    const emxArray_int32_T rows, const int32_T b_rows, emxArray_creal32_T newIm)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_rows);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = rows.data[orow];
    if (q1 > 2147483646 - offsetH) {
      q1 = MAX_int32_T;
    } else {
      q1 = (offsetH + q1) + 1;
    }
    newIm.data[orow] = expanded.data[q1 - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel91(
    const real32_T cv_im, const real32_T cv_re, const int32_T offsetH,
    emxArray_creal32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[offsetH].re = cv_re;
    c.data[offsetH].im = cv_im;
  }
}

//
//
namespace coder {
void b_conv2(const emxArray_creal32_T *cpu_a, const emxArray_real32_T *cpu_b,
             emxArray_creal32_T *cpu_c, boolean_T *c_outdatedOnCpu,
             emxArray_creal32_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal32_T b_gpu_expanded;
  emxArray_creal32_T gpu_a;
  emxArray_creal32_T gpu_newIm;
  emxArray_creal32_T *cpu_expanded;
  emxArray_creal32_T *cpu_newIm;
  emxArray_int32_T b_gpu_rows;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_rows;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real32_T gpu_b;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *b_cpu_expanded;
  nvtxRangePushA("#fcn#b_conv2#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_b);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_a);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_expanded, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_expanded, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &hh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &hh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_rows, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_newIm, 1, &dh_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    int32_T k;
    k = cpu_c->size[0];
    cpu_c->size[0] = 1;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_c, k, &eh_emlrtRTEI);
    cpu_c->data[0].re = cpu_b->data[0] * cpu_a->data[0].re;
    cpu_c->data[0].im = cpu_b->data[0] * cpu_a->data[0].im;
    *c_outdatedOnCpu = false;
    *c_outdatedOnGpu = true;
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    k = cpu_c->size[0];
    cpu_c->size[0] = mc;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_c, k, &ih_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2_kernel81#" MW_AT_LINE);
      conv2_kernel81<<<grid, block>>>(mc - 1, *gpu_c);
    }
    *c_outdatedOnGpu = false;
    *c_outdatedOnCpu = true;
    if (cpu_a->size[0] != 0) {
      if ((cpu_b->size[0] == 1) && (cpu_a->size[0] == 1)) {
        creal32_T a;
        real32_T cv_im;
        real32_T cv_re;
        a = cpu_a->data[0];
        k = cpu_c->size[0];
        cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(cpu_c, k, &ih_emlrtRTEI);
        cv_im = cpu_b->data[0] * a.re;
        cv_re = cpu_b->data[0] * a.im;
        cpu_c->data[0].im = cv_re;
        cpu_c->data[0].re = cv_im;
        *c_outdatedOnCpu = false;
        *c_outdatedOnGpu = true;
      } else {
        real_T blockDims_idx_0;
        int32_T b_OH;
        int32_T n;
        int32_T offsetH;
        uint32_T OH;
        int8_T threadDims_idx_0;
        OH = (static_cast<uint32_T>(cpu_b->size[0]) +
              static_cast<uint32_T>(cpu_a->size[0])) -
             1U;
        mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
        blockDims_idx_0 = std::floor(
            (static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
            static_cast<real_T>(mc));
        threadDims_idx_0 = static_cast<int8_T>(mc);
        if (OH > 2147483647U) {
          OH = 2147483647U;
        }
        b_OH = static_cast<int32_T>(OH);
        offsetH = static_cast<int32_T>(
                      std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                  static_cast<int32_T>(std::floor(
                      (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
        if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
          mc = MAX_int32_T;
        } else {
          mc = static_cast<int32_T>(OH) + cpu_a->size[0];
        }
        k = b_cpu_expanded->size[0];
        b_cpu_expanded->size[0] = mc - 1;
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_cpu_expanded, k, &dh_emlrtRTEI);
        k = (static_cast<int32_T>(OH) + cpu_a->size[0]) - 2;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(b_cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel82#" MW_AT_LINE);
          conv2_kernel82<<<grid, block>>>(k, gpu_expanded);
        }
        n = cpu_b->size[0];
        k = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_b->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(b_cpu_y, k, &fh_emlrtRTEI);
        b_cpu_y->data[0] = 1;
        mc = 1;
        profileLoopStart("b_conv2_loop_1", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          b_cpu_y->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_iv1->size[0];
        cpu_iv1->size[0] = b_cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv1, k, &dh_emlrtRTEI);
        k = b_cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(b_cpu_y, &gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, b_cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel83#" MW_AT_LINE);
          conv2_kernel83<<<grid, block>>>(offsetH, gpu_y, k, gpu_iv1);
        }
        mc = cpu_iv1->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_b, &gpu_b, false);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_b, cpu_b);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel84#" MW_AT_LINE);
          conv2_kernel84<<<grid, block>>>(gpu_b, gpu_iv1, mc - 1, gpu_expanded);
        }
        n = cpu_a->size[0];
        k = b_cpu_rows->size[0] * b_cpu_rows->size[1];
        b_cpu_rows->size[0] = 1;
        b_cpu_rows->size[1] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(b_cpu_rows, k, &fh_emlrtRTEI);
        b_cpu_rows->data[0] = 0;
        mc = 0;
        profileLoopStart("b_conv2_loop_3", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          b_cpu_rows->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_c->size[0];
        cpu_c->size[0] = static_cast<int32_T>(OH);
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(cpu_c, k, &dh_emlrtRTEI);
        if (blockDims_idx_0 < 4.294967296E+9) {
          OH = static_cast<uint32_T>(blockDims_idx_0);
        } else {
          OH = MAX_uint32_T;
        }
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1), dim3(OH, 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid,
            &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(b_cpu_rows, &gpu_rows, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, b_cpu_rows);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel85#" MW_AT_LINE);
          conv2_kernel85<<<grid, block>>>(gpu_expanded, gpu_rows, gpu_a,
                                          b_OH - 1, *gpu_c, cpu_a->size[0U]);
        }
      }
    }
  } else {
    int32_T k;
    int32_T mc;
    boolean_T validLaunchParams;
    if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
      mc = cpu_a->size[0] + cpu_b->size[0];
    } else {
      mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
    }
    k = cpu_c->size[0];
    cpu_c->size[0] = mc;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_c, k, &ah_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2_kernel86#" MW_AT_LINE);
      conv2_kernel86<<<grid, block>>>(mc - 1, *gpu_c);
    }
    *c_outdatedOnGpu = false;
    *c_outdatedOnCpu = true;
    if ((cpu_a->size[0] != 0) && (cpu_b->size[0] != 0)) {
      if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
        real32_T cv_im;
        real32_T cv_re;
        cv_re = cpu_b->data[0];
        k = cpu_c->size[0];
        cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(cpu_c, k, &ah_emlrtRTEI);
        cv_im = cv_re * cpu_a->data[0].re;
        cv_re *= cpu_a->data[0].im;
        cpu_c->data[0].im = cv_re;
        cpu_c->data[0].re = cv_im;
        *c_outdatedOnCpu = false;
        *c_outdatedOnGpu = true;
      } else {
        int32_T b_OH;
        int32_T n;
        int32_T offsetH;
        uint32_T OH;
        boolean_T c_needsGpuEnsureCapacity;
        boolean_T rows_needsGpuEnsureCapacity;
        boolean_T rows_outdatedOnGpu;
        OH = (static_cast<uint32_T>(cpu_a->size[0]) +
              static_cast<uint32_T>(cpu_b->size[0])) -
             1U;
        if (OH > 2147483647U) {
          OH = 2147483647U;
        }
        b_OH = static_cast<int32_T>(OH);
        offsetH = static_cast<int32_T>(
                      std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                  static_cast<int32_T>(std::floor(
                      (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
        if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
          mc = MAX_int32_T;
        } else {
          mc = static_cast<int32_T>(OH) + cpu_b->size[0];
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(cpu_expanded, k, &dh_emlrtRTEI);
        k = (static_cast<int32_T>(OH) + cpu_b->size[0]) - 2;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(cpu_expanded, &b_gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel87#" MW_AT_LINE);
          conv2_kernel87<<<grid, block>>>(k, b_gpu_expanded);
        }
        n = cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_y, k, &fh_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        profileLoopStart("b_conv2_loop_0", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, k, &dh_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &b_gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel88#" MW_AT_LINE);
          conv2_kernel88<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv);
        }
        mc = cpu_iv->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel89#" MW_AT_LINE);
          conv2_kernel89<<<grid, block>>>(gpu_a, gpu_iv, mc - 1,
                                          b_gpu_expanded);
        }
        n = cpu_b->size[0];
        k = cpu_rows->size[0] * cpu_rows->size[1];
        cpu_rows->size[0] = 1;
        cpu_rows->size[1] = cpu_b->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_rows, k, &fh_emlrtRTEI);
        cpu_rows->data[0] = 0;
        rows_outdatedOnGpu = true;
        rows_needsGpuEnsureCapacity = true;
        mc = 0;
        profileLoopStart("b_conv2_loop_2", __LINE__, (n - 2) + 1, "");
        for (k = 0; k <= n - 2; k++) {
          mc++;
          cpu_rows->data[k + 1] = mc;
        }
        profileLoopEnd();
        k = cpu_c->size[0];
        cpu_c->size[0] = static_cast<int32_T>(OH);
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(cpu_c, k, &dh_emlrtRTEI);
        c_needsGpuEnsureCapacity = true;
        mc = cpu_b->size[0];
        profileLoopStart("b_conv2_loop_4", __LINE__, (b_OH - 1) + 1, "");
        for (offsetH = 0; offsetH < b_OH; offsetH++) {
          real32_T cv_im;
          real32_T cv_re;
          boolean_T newIm_outdatedOnCpu;
          k = cpu_newIm->size[0];
          cpu_newIm->size[0] = cpu_rows->size[1];
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(cpu_newIm, k, &dh_emlrtRTEI);
          k = cpu_rows->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(k), &grid, &block,
                                  2147483647U);
          if (rows_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_int32_T(cpu_rows, &b_gpu_rows,
                                         !rows_outdatedOnGpu);
          }
          rows_needsGpuEnsureCapacity = false;
          nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal32_T(cpu_newIm, &gpu_newIm, true);
          if (rows_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_rows, cpu_rows);
          }
          rows_outdatedOnGpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#conv2_kernel90#" MW_AT_LINE);
            conv2_kernel90<<<grid, block>>>(b_gpu_expanded, offsetH, b_gpu_rows,
                                            k, gpu_newIm);
          }
          newIm_outdatedOnCpu = true;
          cv_re = 0.0F;
          cv_im = 0.0F;
          profileLoopStart("b_conv2_loop_5", __LINE__, (mc - 1) + 1, "");
          for (n = 0; n < mc; n++) {
            if (newIm_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_creal32_T(cpu_newIm, &gpu_newIm);
            }
            cv_re +=
                cpu_newIm->data[n].re * cpu_b->data[(cpu_b->size[0] - n) - 1];
            newIm_outdatedOnCpu = false;
            cv_im +=
                cpu_newIm->data[n].im * cpu_b->data[(cpu_b->size[0] - n) - 1];
          }
          profileLoopEnd();
          if (c_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
          }
          c_needsGpuEnsureCapacity = false;
          nvtxMarkA("#conv2_kernel91#" MW_AT_LINE);
          conv2_kernel91<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              cv_im, cv_re, offsetH, *gpu_c);
        }
        profileLoopEnd();
      }
    }
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_newIm);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_expanded);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_b);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_rows);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_newIm);
  nvtxRangePop();
}

//
//
void conv2(const emxArray_creal32_T *cpu_a, emxArray_real32_T *cpu_b,
           boolean_T *b_outdatedOnCpu, emxArray_real32_T *gpu_b,
           boolean_T *b_outdatedOnGpu, emxArray_creal32_T *cpu_c,
           boolean_T *c_outdatedOnCpu, emxArray_creal32_T *gpu_c,
           boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal32_T b_gpu_a;
  emxArray_creal32_T b_gpu_c;
  emxArray_creal32_T b_gpu_expanded;
  emxArray_creal32_T c_gpu_a;
  emxArray_creal32_T gpu_a;
  emxArray_creal32_T gpu_expanded;
  emxArray_creal32_T gpu_newIm;
  emxArray_creal32_T *b_cpu_a;
  emxArray_creal32_T *b_cpu_c;
  emxArray_creal32_T *b_cpu_expanded;
  emxArray_creal32_T *c_cpu_a;
  emxArray_creal32_T *cpu_expanded;
  emxArray_creal32_T *cpu_newIm;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T c_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_iv2;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *c_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_iv2;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  creal32_T(*gpu_b_data)[1];
  real32_T *gpu_cv_re;
  real32_T *gpu_f;
  boolean_T a_outdatedOnGpu;
  nvtxRangePushA("#fcn#conv2#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_cv_re, 4UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_f, 4UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_b_data, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&c_gpu_a);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv2);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxReset_int32_T#" MW_AT_LINE);
  gpuEmxReset_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_a);
  a_outdatedOnGpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&b_cpu_c, 1, &gh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&b_cpu_a, 1, &xg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_expanded, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_y, 2, &hh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&b_cpu_expanded, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&b_cpu_y, 2, &hh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv1, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&c_cpu_y, 2, &hh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_iv2, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&cpu_rows, 2, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_newIm, 1, &dh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&c_cpu_a, 1, &ch_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    if (cpu_a->size[1] < 1) {
      int32_T i;
      i = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(cpu_c, i, &yg_emlrtRTEI);
      cpu_c->data[0].im = 0.0F;
      cpu_c->data[0].re = 0.0F;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      int32_T i;
      real32_T cv_im;
      boolean_T validLaunchParams;
      i = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(b_cpu_a, i, &xg_emlrtRTEI);
      i = cpu_a->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(b_cpu_a, &b_gpu_a, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel52#" MW_AT_LINE);
        conv2_kernel52<<<grid, block>>>(gpu_a, i, b_gpu_a, cpu_a->size[0U]);
      }
      if (*b_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      cv_im = cpu_b->data[0];
      i = b_cpu_c->size[0];
      b_cpu_c->size[0] = b_cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(b_cpu_c, i, &bh_emlrtRTEI);
      i = b_cpu_a->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(b_cpu_c, &b_gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel53#" MW_AT_LINE);
        conv2_kernel53<<<grid, block>>>(i, b_gpu_c);
      }
      if (b_cpu_a->size[0] == 1) {
        cv_im = cpu_b->data[0];
        i = b_cpu_c->size[0];
        b_cpu_c->size[0] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(b_cpu_c, i, &bh_emlrtRTEI);
        nvtxMarkA("#conv2_kernel54#" MW_AT_LINE);
        conv2_kernel54<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            b_gpu_a, cv_im, gpu_f, gpu_cv_re);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(b_cpu_c, &b_gpu_c, true);
        nvtxMarkA("#conv2_kernel55#" MW_AT_LINE);
        conv2_kernel55<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_cv_re,
                                                                gpu_f, b_gpu_c);
      } else {
        int32_T b_OH;
        int32_T mc;
        int32_T nc;
        int32_T offsetH;
        int8_T threadDims_idx_0;
        nc = static_cast<int32_T>(std::fmin(
            32.0, (static_cast<real_T>(b_cpu_a->size[0]) + 1.0) - 1.0));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(b_cpu_a->size[0]) +
                                 static_cast<uint32_T>(nc)) -
             1.0) /
            static_cast<real_T>(nc)));
        threadDims_idx_0 = static_cast<int8_T>(nc);
        b_OH = b_cpu_a->size[0];
        if (b_cpu_a->size[0] > 2147483646) {
          mc = MAX_int32_T;
        } else {
          mc = b_cpu_a->size[0] + 1;
        }
        i = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(cpu_expanded, i, &dh_emlrtRTEI);
        i = b_cpu_a->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel57#" MW_AT_LINE);
          conv2_kernel57<<<grid, block>>>(i, gpu_expanded);
        }
        nc = b_cpu_a->size[0];
        i = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = b_cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_y, i, &fh_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        profileLoopStart("conv2_loop_0", __LINE__, (nc - 2) + 1, "");
        for (int32_T k{0}; k <= nc - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        profileLoopEnd();
        i = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
        emxEnsureCapacity_int32_T(cpu_iv, i, &dh_emlrtRTEI);
        i = cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel58#" MW_AT_LINE);
          conv2_kernel58<<<grid, block>>>(gpu_y, i, gpu_iv);
        }
        mc = cpu_iv->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel59#" MW_AT_LINE);
          conv2_kernel59<<<grid, block>>>(b_gpu_a, gpu_iv, mc - 1,
                                          gpu_expanded);
        }
        i = b_cpu_c->size[0];
        b_cpu_c->size[0] = b_cpu_a->size[0];
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(b_cpu_c, i, &dh_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid,
            &block);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(b_cpu_c, &b_gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#conv2_kernel60#" MW_AT_LINE);
          conv2_kernel60<<<grid, block>>>(gpu_expanded, cv_im, b_OH - 1,
                                          b_gpu_c);
        }
      }
      i = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = b_cpu_c->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(cpu_c, i, &eh_emlrtRTEI);
      i = b_cpu_c->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel56#" MW_AT_LINE);
        conv2_kernel56<<<grid, block>>>(b_gpu_c, i, *gpu_c, cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    nvtxMarkA("#conv2NonSeparable#" MW_AT_LINE);
    conv2NonSeparable(cpu_b, b_outdatedOnCpu, gpu_b, b_outdatedOnGpu,
                      (emxArray_creal32_T *)cpu_a, &gpu_a, &a_outdatedOnGpu,
                      cpu_c, c_outdatedOnCpu, gpu_c, c_outdatedOnGpu);
  } else {
    int32_T i;
    int32_T mc;
    int32_T nc;
    boolean_T validLaunchParams;
    if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
      mc = cpu_a->size[0] + cpu_b->size[0];
    } else {
      mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
    }
    if (cpu_a->size[1] == 0) {
      nc = 1;
    } else {
      nc = cpu_a->size[1];
    }
    i = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_c, i, &ah_emlrtRTEI);
    i = mc * nc - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#conv2_kernel61#" MW_AT_LINE);
      conv2_kernel61<<<grid, block>>>(i, *gpu_c);
    }
    *c_outdatedOnGpu = false;
    *c_outdatedOnCpu = true;
    if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0) ||
        (cpu_b->size[0] == 0)) {
      i = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = nc;
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(cpu_c, i, &ah_emlrtRTEI);
      i = mc * nc - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel62#" MW_AT_LINE);
        conv2_kernel62<<<grid, block>>>(i, *gpu_c);
      }
    } else if ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1) &&
               (cpu_b->size[0] == 1)) {
      real32_T cpu_cv_re;
      if (*b_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      cpu_cv_re = cpu_b->data[0];
      nvtxMarkA("#conv2_kernel63#" MW_AT_LINE);
      conv2_kernel63<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(cpu_cv_re,
                                                              *gpu_b_data);
      i = c_cpu_a->size[0];
      c_cpu_a->size[0] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(c_cpu_a, i, &ch_emlrtRTEI);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(c_cpu_a, &c_gpu_a, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
      nvtxMarkA("#conv2_kernel64#" MW_AT_LINE);
      conv2_kernel64<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          gpu_a, *gpu_b_data, c_gpu_a);
      i = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = nc;
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(cpu_c, i, &ah_emlrtRTEI);
      i = mc * nc - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel65#" MW_AT_LINE);
        conv2_kernel65<<<grid, block>>>(c_gpu_a, i, *gpu_c);
      }
    } else {
      int32_T iv1[2];
      int32_T OW;
      int32_T b_OH;
      int32_T offsetH;
      uint32_T OH;
      boolean_T a_outdatedOnCpu;
      boolean_T c_needsGpuEnsureCapacity;
      OH = (static_cast<uint32_T>(cpu_a->size[0]) +
            static_cast<uint32_T>(cpu_b->size[0])) -
           1U;
      OW = cpu_a->size[1];
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
      if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_b->size[0];
      }
      i = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[0] = mc - 1;
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(b_cpu_expanded, i, &dh_emlrtRTEI);
      if (cpu_a->size[1] > 2147483646) {
        mc = MAX_int32_T;
      } else {
        mc = cpu_a->size[1] + 1;
      }
      i = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[1] = mc - 1;
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(b_cpu_expanded, i, &dh_emlrtRTEI);
      i = ((static_cast<int32_T>(OH) + cpu_b->size[0]) - 1) * cpu_a->size[1] -
          1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(b_cpu_expanded, &b_gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel66#" MW_AT_LINE);
        conv2_kernel66<<<grid, block>>>(i, b_gpu_expanded);
      }
      nc = cpu_a->size[0];
      i = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = 1;
      b_cpu_y->size[1] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(b_cpu_y, i, &fh_emlrtRTEI);
      b_cpu_y->data[0] = 1;
      mc = 1;
      profileLoopStart("conv2_loop_1", __LINE__, (nc - 2) + 1, "");
      for (int32_T k{0}; k <= nc - 2; k++) {
        mc++;
        b_cpu_y->data[k + 1] = mc;
      }
      profileLoopEnd();
      i = cpu_iv1->size[0];
      cpu_iv1->size[0] = b_cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_iv1, i, &dh_emlrtRTEI);
      i = b_cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel67#" MW_AT_LINE);
        conv2_kernel67<<<grid, block>>>(offsetH, b_gpu_y, i, gpu_iv1);
      }
      nc = cpu_a->size[1];
      i = c_cpu_y->size[0] * c_cpu_y->size[1];
      c_cpu_y->size[0] = 1;
      c_cpu_y->size[1] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(c_cpu_y, i, &fh_emlrtRTEI);
      c_cpu_y->data[0] = 1;
      mc = 1;
      profileLoopStart("conv2_loop_2", __LINE__, (nc - 2) + 1, "");
      for (int32_T k{0}; k <= nc - 2; k++) {
        mc++;
        c_cpu_y->data[k + 1] = mc;
      }
      profileLoopEnd();
      i = cpu_iv2->size[0];
      cpu_iv2->size[0] = c_cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_iv2, i, &dh_emlrtRTEI);
      i = c_cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(c_cpu_y, &c_gpu_y, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_int32_T(cpu_iv2, &gpu_iv2, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_int32_T(&c_gpu_y, c_cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel68#" MW_AT_LINE);
        conv2_kernel68<<<grid, block>>>(c_gpu_y, i, gpu_iv2);
      }
      iv1[0] = cpu_iv1->size[0];
      iv1[1] = cpu_iv2->size[0];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(iv1[1] - 1, iv1[0] - 1), &grid,
                              &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#conv2_kernel69#" MW_AT_LINE);
        conv2_kernel69<<<grid, block>>>(gpu_a, iv1[0], gpu_iv2, gpu_iv1,
                                        iv1[0] - 1, iv1[1] - 1, b_gpu_expanded,
                                        b_cpu_expanded->size[0U]);
      }
      nc = cpu_b->size[0];
      i = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_b->size[0];
      nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
      emxEnsureCapacity_int32_T(cpu_rows, i, &fh_emlrtRTEI);
      cpu_rows->data[0] = 0;
      a_outdatedOnCpu = true;
      a_outdatedOnGpu = true;
      mc = 0;
      profileLoopStart("conv2_loop_3", __LINE__, (nc - 2) + 1, "");
      for (int32_T k{0}; k <= nc - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      profileLoopEnd();
      i = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      cpu_c->size[1] = cpu_a->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(cpu_c, i, &dh_emlrtRTEI);
      c_needsGpuEnsureCapacity = true;
      mc = cpu_b->size[0];
      profileLoopStart("conv2_loop_4", __LINE__, (OW - 1) + 1, "");
      for (offsetH = 0; offsetH < OW; offsetH++) {
        profileLoopStart("conv2_loop_5", __LINE__, (b_OH - 1) + 1, "");
        for (int32_T k{0}; k < b_OH; k++) {
          real32_T cpu_cv_re;
          real32_T cv_im;
          boolean_T newIm_outdatedOnCpu;
          i = cpu_newIm->size[0];
          cpu_newIm->size[0] = cpu_rows->size[1];
          nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
          emxEnsureCapacity_creal32_T(cpu_newIm, i, &dh_emlrtRTEI);
          i = cpu_rows->size[1] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                  2147483647U);
          if (a_outdatedOnGpu) {
            nvtxMarkA("#gpuEmxEnsureCapacity_int32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, !a_outdatedOnCpu);
          }
          a_outdatedOnGpu = false;
          nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal32_T(cpu_newIm, &gpu_newIm, true);
          if (a_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LINE);
            gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
          }
          a_outdatedOnCpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#conv2_kernel70#" MW_AT_LINE);
            conv2_kernel70<<<grid, block>>>(offsetH, b_gpu_expanded, k,
                                            gpu_rows, i, gpu_newIm,
                                            b_cpu_expanded->size[0U]);
          }
          newIm_outdatedOnCpu = true;
          cpu_cv_re = 0.0F;
          cv_im = 0.0F;
          profileLoopStart("conv2_loop_6", __LINE__, (mc - 1) + 1, "");
          for (nc = 0; nc < mc; nc++) {
            if (newIm_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_creal32_T(cpu_newIm, &gpu_newIm);
            }
            if (*b_outdatedOnCpu) {
              nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
            }
            cpu_cv_re +=
                cpu_newIm->data[nc].re * cpu_b->data[(cpu_b->size[0] - nc) - 1];
            newIm_outdatedOnCpu = false;
            *b_outdatedOnCpu = false;
            cv_im +=
                cpu_newIm->data[nc].im * cpu_b->data[(cpu_b->size[0] - nc) - 1];
          }
          profileLoopEnd();
          if (c_needsGpuEnsureCapacity) {
            nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
            gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
          }
          nvtxMarkA("#conv2_kernel71#" MW_AT_LINE);
          conv2_kernel71<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              cpu_cv_re, offsetH, k, *gpu_c, cpu_c->size[0U]);
          c_needsGpuEnsureCapacity = false;
          nvtxMarkA("#conv2_kernel72#" MW_AT_LINE);
          conv2_kernel72<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              cv_im, offsetH, k, *gpu_c, cpu_c->size[0U]);
        }
        profileLoopEnd();
      }
      profileLoopEnd();
    }
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&c_cpu_a);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_newIm);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_rows);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv2);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&c_cpu_y);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv1);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_iv);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&b_cpu_expanded);
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&cpu_y);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_expanded);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&b_cpu_a);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_a);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&b_gpu_a);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&b_gpu_expanded);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv1);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_iv2);
  nvtxMarkA("#gpuEmxFree_int32_T#" MW_AT_LINE);
  gpuEmxFree_int32_T(&gpu_rows);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_newIm);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&c_gpu_a);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_b_data), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_f), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_cv_re), __FILE__, __LINE__);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (conv2.cu)

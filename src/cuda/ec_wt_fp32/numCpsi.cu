#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// numCpsi.cu
//
// Code generation for function 'numCpsi'
//

// Include files
#include "numCpsi.h"
#include "cospiAndSinpi.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "quadgk.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo be_emlrtRTEI{
    28,        // lineNo
    47,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo ce_emlrtRTEI{
    28,        // lineNo
    36,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo de_emlrtRTEI{
    28,        // lineNo
    23,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo ee_emlrtRTEI{
    28,        // lineNo
    46,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo fe_emlrtRTEI{
    1,         // lineNo
    17,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

static emlrtRTEInfo ge_emlrtRTEI{
    28,        // lineNo
    19,        // colNo
    "numCpsi", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "numCpsi.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
real_T numCpsi(const char_T wname[5], real_T varargin_1, real_T varargin_2)
{
  static const real_T gam[23]{1.0,
                              1.0,
                              2.0,
                              6.0,
                              24.0,
                              120.0,
                              720.0,
                              5040.0,
                              40320.0,
                              362880.0,
                              3.6288E+6,
                              3.99168E+7,
                              4.790016E+8,
                              6.2270208E+9,
                              8.71782912E+10,
                              1.307674368E+12,
                              2.0922789888E+13,
                              3.55687428096E+14,
                              6.402373705728E+15,
                              1.21645100408832E+17,
                              2.43290200817664E+18,
                              5.109094217170944E+19,
                              1.1240007277776077E+21};
  static const real_T p[8]{-1.716185138865495,  24.76565080557592,
                           -379.80425647094563, 629.3311553128184,
                           866.96620279041326,  -31451.272968848367,
                           -36144.413418691176, 66456.143820240541};
  static const real_T q[8]{-30.840230011973897, 315.35062697960416,
                           -1015.1563674902192, -3107.7716715723109,
                           22538.11842098015,   4755.8462775278813,
                           -134659.95986496931, -115132.25967555349};
  static const char_T cv1[5]{'m', 'o', 'r', 's', 'e'};
  static const char_T vstr[5]{'m', 'o', 'r', 's', 'e'};
  emxArray_real_T *a;
  emxArray_real_T *b_a;
  emxArray_real_T *b_x;
  emxArray_real_T *c_x;
  emxArray_real_T *d_x;
  emxArray_real_T *e_x;
  emxArray_real_T *f_x;
  emxArray_real_T *fx;
  emxArray_real_T *xt;
  emxArray_real_T *y;
  real_T interval[650];
  real_T absxk;
  real_T anorm;
  real_T cpsi;
  real_T pathlen;
  int32_T exitg1;
  int32_T ix;
  int32_T nx;
  char_T partial_match[5];
  char_T wavname[5];
  boolean_T first_iteration;
  nvtxRangePushA("#fcn#numCpsi#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nx = 0;
  first_iteration = false;
  ix = 0;
  nvtxRangePushA("#loop#numCpsi_whileloop_3##" MW_AT_LINE);
  do {
    exitg1 = 0;
    if (ix + 1 < 6) {
      if (cv[static_cast<uint8_T>(wname[ix]) & 127] !=
          cv[static_cast<int32_T>(cv1[ix])]) {
        exitg1 = 1;
      } else {
        ix++;
      }
    } else {
      first_iteration = true;
      exitg1 = 1;
    }
  } while (exitg1 == 0);
  nvtxRangePop();
  if (first_iteration) {
    nx = 1;
    profileLoopStart("numCpsi_loop_1", __LINE__, 4 + 1, "");
    for (ix = 0; ix < 5; ix++) {
      partial_match[ix] = vstr[ix];
    }
    profileLoopEnd();
  } else {
    profileLoopStart("numCpsi_loop_0", __LINE__, 4 + 1, "");
    for (ix = 0; ix < 5; ix++) {
      partial_match[ix] = ' ';
    }
    profileLoopEnd();
  }
  if (nx != 0) {
    profileLoopStart("numCpsi_loop_2", __LINE__, 4 + 1, "");
    for (ix = 0; ix < 5; ix++) {
      wavname[ix] = partial_match[ix];
    }
    profileLoopEnd();
  }
  anorm = 2.0 * std::exp(varargin_2 / varargin_1 *
                         ((std::log(varargin_1) - std::log(varargin_2)) + 1.0));
  first_iteration = false;
  ix = 0;
  nvtxRangePushA("#loop#numCpsi_whileloop_2##" MW_AT_LINE);
  do {
    exitg1 = 0;
    if (ix + 1 < 6) {
      if (cv[static_cast<int32_T>(wavname[ix])] !=
          cv[static_cast<int32_T>(cv1[ix])]) {
        exitg1 = 1;
      } else {
        ix++;
      }
    } else {
      first_iteration = true;
      exitg1 = 1;
    }
  } while (exitg1 == 0);
  nvtxRangePop();
  if (first_iteration) {
    real_T x;
    x = 2.0 * varargin_2 / varargin_1;
    if ((x >= 1.0) && (x <= 23.0) && (x == std::floor(x))) {
      x = gam[static_cast<int32_T>(x) - 1];
    } else if ((x < 1.0) && (x == std::floor(x))) {
      x = rtInf;
    } else if (!std::isnan(x)) {
      if (std::isinf(x)) {
        x = rtInf;
      } else {
        real_T midpt;
        real_T tol;
        tol = 1.0;
        nx = 1;
        first_iteration = false;
        if (x <= 0.0) {
          midpt = std::floor(-x);
          first_iteration = (midpt != std::floor(-x / 2.0) * 2.0);
          nvtxMarkA("#cospiAndSinpi#" MW_AT_LINE);
          ::coder::internal::scalar::cospiAndSinpi(-x - midpt, &absxk);
          tol = -3.1415926535897931 / absxk;
          x = -x + 1.0;
        }
        if (x < 12.0) {
          real_T abserrsubk;
          real_T halfh;
          abserrsubk = x;
          if (x < 1.0) {
            midpt = x;
            x++;
          } else {
            nx = static_cast<int32_T>(std::floor(x));
            x -= std::floor(x) - 1.0;
            midpt = x - 1.0;
          }
          absxk = 0.0 * midpt;
          halfh = 1.0;
          profileLoopStart("numCpsi_loop_5", __LINE__, 7 + 1, "");
          for (ix = 0; ix < 8; ix++) {
            absxk = (absxk + p[ix]) * midpt;
            halfh = halfh * midpt + q[ix];
          }
          profileLoopEnd();
          midpt = absxk / halfh + 1.0;
          if (abserrsubk < x) {
            midpt /= abserrsubk;
          } else if (abserrsubk > x) {
            profileLoopStart("numCpsi_loop_6", __LINE__, (nx - 2) + 1, "");
            for (int32_T j{0}; j <= nx - 2; j++) {
              midpt *= x;
              x++;
            }
            profileLoopEnd();
          }
        } else {
          real_T halfh;
          midpt = x * x;
          halfh = 0.0057083835261;
          profileLoopStart("numCpsi_loop_4", __LINE__, 5 + 1, "");
          for (ix = 0; ix < 6; ix++) {
            halfh = halfh / midpt + dv3[ix];
          }
          profileLoopEnd();
          halfh = (halfh / x - x) + 0.91893853320467278;
          halfh += (x - 0.5) * std::log(x);
          midpt = std::exp(halfh);
        }
        if (first_iteration) {
          midpt = -midpt;
        }
        if (tol != 1.0) {
          midpt = tol / midpt;
        }
        x = midpt;
      }
    }
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    cpsi = anorm * anorm / (2.0 * varargin_1) *
           rt_powd_snf(0.5, 2.0 * (varargin_2 / varargin_1) - 1.0) * x;
  } else {
    real_T subs[1298];
    real_T errsub[649];
    real_T qsub[649];
    interval[0] = -1.0;
    interval[1] = 1.0;
    std::memset(&interval[2], 0, 648U * sizeof(real_T));
    cpsi = 0.0;
    nvtxMarkA("#split#" MW_AT_LINE);
    ix = split(interval, 2, &pathlen);
    if (!(pathlen > 0.0)) {
      cpsi = 0.47999999999999976;
    } else {
      real_T err_ok;
      real_T q_ok;
      int32_T k;
      int32_T nsubs;
      nsubs = ix - 2;
      profileLoopStart("numCpsi_loop_3", __LINE__, nsubs + 1, "");
      for (k = 0; k <= nsubs; k++) {
        subs[k << 1] = interval[k];
        subs[(k << 1) + 1] = interval[k + 1];
      }
      profileLoopEnd();
      q_ok = 0.0;
      err_ok = 0.0;
      first_iteration = true;
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&b_x, 2, &lc_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&c_x, 2, &rc_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&xt, 2, &sc_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&y, 2, &ee_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&a, 2, &be_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&fx, 2, &fe_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&d_x, 2, &ce_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&b_a, 2, &be_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&e_x, 2, &de_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&f_x, 2, &ge_emlrtRTEI, true);
      nvtxRangePushA("#loop#numCpsi_whileloop_0##" MW_AT_LINE);
      do {
        real_T halfh;
        real_T midpt;
        boolean_T guard1;
        exitg1 = 0;
        ix = b_x->size[0] * b_x->size[1];
        b_x->size[0] = 1;
        b_x->size[1] = 15 * (nsubs + 1);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_x, ix, &lc_emlrtRTEI);
        ix = -1;
        profileLoopStart("numCpsi_loop_7", __LINE__, nsubs + 1, "");
        for (k = 0; k <= nsubs; k++) {
          anorm = subs[k << 1];
          absxk = subs[(k << 1) + 1];
          midpt = (anorm + absxk) / 2.0;
          halfh = (absxk - anorm) / 2.0;
          profileLoopStart("numCpsi_loop_8", __LINE__, 14 + 1, "");
          for (int32_T j{0}; j < 15; j++) {
            b_x->data[(ix + j) + 1] = dv[j] * halfh + midpt;
          }
          profileLoopEnd();
          ix += 15;
        }
        profileLoopEnd();
        ix = c_x->size[0] * c_x->size[1];
        c_x->size[0] = 1;
        c_x->size[1] = b_x->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_x, ix, &mc_emlrtRTEI);
        ix = xt->size[0] * xt->size[1];
        xt->size[0] = 1;
        xt->size[1] = b_x->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(xt, ix, &nc_emlrtRTEI);
        ix = b_x->size[1];
        profileLoopStart("numCpsi_loop_9", __LINE__, (ix - 1) + 1, "");
        for (k = 0; k < ix; k++) {
          anorm = b_x->data[k];
          midpt = anorm * anorm;
          c_x->data[k] = 0.29999999999999982 * anorm * (3.0 - midpt) + 5.0;
          xt->data[k] = 0.89999999999999947 * (1.0 - midpt);
        }
        profileLoopEnd();
        guard1 = false;
        if (!first_iteration) {
          boolean_T exitg2;
          absxk = std::abs(c_x->data[0]);
          k = 0;
          exitg2 = false;
          nvtxRangePushA("#loop#numCpsi_whileloop_1##" MW_AT_LINE);
          while ((!exitg2) && (k <= c_x->size[1] - 2)) {
            real_T x;
            x = absxk;
            absxk = std::abs(c_x->data[k + 1]);
            if (std::abs(c_x->data[k + 1] - c_x->data[k]) <=
                2.2204460492503131E-14 * std::fmax(x, absxk)) {
              first_iteration = true;
              exitg2 = true;
            } else {
              k++;
            }
          }
          nvtxRangePop();
          if (first_iteration) {
            int16_T dv_idx_1;
            dv_idx_1 = static_cast<int16_T>(b_x->size[1]);
            ix = fx->size[0] * fx->size[1];
            fx->size[0] = 1;
            fx->size[1] = b_x->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(fx, ix, &pc_emlrtRTEI);
            profileLoopStart("numCpsi_loop_13", __LINE__, (dv_idx_1 - 1) + 1,
                             "");
            for (ix = 0; ix < dv_idx_1; ix++) {
              fx->data[ix] = 0.0;
            }
            profileLoopEnd();
          } else {
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1) {
          first_iteration = false;
          ix = a->size[0] * a->size[1];
          a->size[0] = 1;
          a->size[1] = c_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(a, ix, &be_emlrtRTEI);
          profileLoopStart("numCpsi_loop_11", __LINE__, (c_x->size[1] - 1) + 1,
                           "");
          for (ix = 0; ix < c_x->size[1]; ix++) {
            a->data[ix] = (c_x->data[ix] - 5.0) / 0.6;
          }
          profileLoopEnd();
          ix = y->size[0] * y->size[1];
          y->size[0] = 1;
          y->size[1] = a->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(y, ix, &m_emlrtRTEI);
          nx = a->size[1];
          profileLoopStart("numCpsi_loop_15", __LINE__, (nx - 1) + 1, "");
          for (k = 0; k < nx; k++) {
            anorm = a->data[k];
            y->data[k] = anorm * anorm;
          }
          profileLoopEnd();
          ix = d_x->size[0] * d_x->size[1];
          d_x->size[0] = 1;
          d_x->size[1] = y->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(d_x, ix, &ce_emlrtRTEI);
          profileLoopStart("numCpsi_loop_17", __LINE__, (y->size[1] - 1) + 1,
                           "");
          for (ix = 0; ix < y->size[1]; ix++) {
            d_x->data[ix] = -2.0 * (1.0 / (1.0 - y->data[ix]));
          }
          profileLoopEnd();
          ix = b_a->size[0] * b_a->size[1];
          b_a->size[0] = 1;
          b_a->size[1] = c_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_a, ix, &be_emlrtRTEI);
          profileLoopStart("numCpsi_loop_18", __LINE__, (c_x->size[1] - 1) + 1,
                           "");
          for (ix = 0; ix < c_x->size[1]; ix++) {
            b_a->data[ix] = (c_x->data[ix] - 5.0) / 0.6;
          }
          profileLoopEnd();
          nx = b_a->size[1];
          profileLoopStart("numCpsi_loop_19", __LINE__, (nx - 1) + 1, "");
          for (k = 0; k < nx; k++) {
            d_x->data[k] = std::exp(d_x->data[k]);
          }
          profileLoopEnd();
          ix = e_x->size[0] * e_x->size[1];
          e_x->size[0] = 1;
          e_x->size[1] = d_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(e_x, ix, &de_emlrtRTEI);
          profileLoopStart("numCpsi_loop_20", __LINE__, (d_x->size[1] - 1) + 1,
                           "");
          for (ix = 0; ix < d_x->size[1]; ix++) {
            e_x->data[ix] = 14.7781121978613 * d_x->data[ix];
          }
          profileLoopEnd();
          nx = d_x->size[1];
          ix = f_x->size[0] * f_x->size[1];
          f_x->size[0] = 1;
          f_x->size[1] = e_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(f_x, ix, &j_emlrtRTEI);
          profileLoopStart("numCpsi_loop_21", __LINE__, (nx - 1) + 1, "");
          for (k = 0; k < nx; k++) {
            f_x->data[k] = std::abs(e_x->data[k]);
          }
          profileLoopEnd();
          if (f_x->size[1] == c_x->size[1]) {
            ix = fx->size[0] * fx->size[1];
            fx->size[0] = 1;
            fx->size[1] = f_x->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(fx, ix, &qc_emlrtRTEI);
            profileLoopStart("numCpsi_loop_22", __LINE__,
                             (f_x->size[1] - 1) + 1, "");
            for (ix = 0; ix < f_x->size[1]; ix++) {
              fx->data[ix] = f_x->data[ix] / c_x->data[ix] * xt->data[ix];
            }
            profileLoopEnd();
          } else {
            nvtxMarkA("#binary_expand_op_7#" MW_AT_LINE);
            binary_expand_op_7(fx, f_x, c_x, xt);
          }
        }
        if (first_iteration) {
          exitg1 = 1;
        } else {
          real_T tol;
          midpt = 0.0;
          ix = -1;
          profileLoopStart("numCpsi_loop_10", __LINE__, nsubs + 1, "");
          for (k = 0; k <= nsubs; k++) {
            anorm = 0.0;
            absxk = 0.0;
            profileLoopStart("numCpsi_loop_12", __LINE__, 14 + 1, "");
            for (int32_T j{0}; j < 15; j++) {
              anorm += dv1[j] * fx->data[(ix + j) + 1];
              absxk += dv2[j] * fx->data[(ix + j) + 1];
            }
            profileLoopEnd();
            ix += 15;
            halfh = (subs[(k << 1) + 1] - subs[k << 1]) / 2.0;
            anorm *= halfh;
            qsub[k] = anorm;
            midpt += anorm;
            errsub[k] = absxk * halfh;
          }
          profileLoopEnd();
          cpsi = midpt + q_ok;
          tol = std::fmax(1.0E-10, 1.0E-6 * std::abs(cpsi));
          absxk = 2.0 * tol / pathlen;
          midpt = 0.0;
          ix = 0;
          profileLoopStart("numCpsi_loop_14", __LINE__, nsubs + 1, "");
          for (k = 0; k <= nsubs; k++) {
            real_T abserrsubk;
            anorm = errsub[k];
            abserrsubk = std::abs(anorm);
            if (abserrsubk <=
                absxk * ((subs[(k << 1) + 1] - subs[k << 1]) / 2.0)) {
              err_ok += anorm;
              q_ok += qsub[k];
            } else {
              midpt += abserrsubk;
              ix++;
              subs[(ix - 1) << 1] = subs[k << 1];
              subs[((ix - 1) << 1) + 1] = subs[(k << 1) + 1];
            }
          }
          profileLoopEnd();
          halfh = std::abs(err_ok) + midpt;
          if ((!std::isinf(cpsi)) && (!std::isnan(cpsi)) &&
              ((!std::isinf(halfh)) && (!std::isnan(halfh))) && (ix != 0) &&
              (!(halfh <= tol))) {
            nsubs = (ix << 1) - 1;
            if (nsubs + 1 > 650) {
              exitg1 = 1;
            } else {
              profileLoopStart("numCpsi_loop_16", __LINE__, (ix - 1) + 1, "");
              for (k = 0; k < ix; k++) {
                subs[((((ix - k) << 1) - 1) << 1) + 1] =
                    subs[(((ix - k) - 1) << 1) + 1];
                subs[(((ix - k) << 1) - 1) << 1] =
                    (subs[((ix - k) - 1) << 1] +
                     subs[(((ix - k) - 1) << 1) + 1]) /
                    2.0;
                subs[((((ix - k) << 1) - 2) << 1) + 1] =
                    subs[(((ix - k) << 1) - 1) << 1];
                subs[(((ix - k) << 1) - 2) << 1] = subs[((ix - k) - 1) << 1];
              }
              profileLoopEnd();
            }
          } else {
            exitg1 = 1;
          }
        }
      } while (exitg1 == 0);
      nvtxRangePop();
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&f_x);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&e_x);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&b_a);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&d_x);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&fx);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&a);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&y);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&xt);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&c_x);
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&b_x);
    }
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
  return cpsi;
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (numCpsi.cu)

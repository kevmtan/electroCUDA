#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// uniformResampleKernel.cu
//
// Code generation for function 'uniformResampleKernel'
//

// Include files
#include "uniformResampleKernel.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "upfirdn.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo kh_emlrtRTEI{
    59,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo lh_emlrtRTEI{
    17,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo mh_emlrtRTEI{
    23,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo nh_emlrtRTEI{
    55,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo oh_emlrtRTEI{
    13,                  // lineNo
    1,                   // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo ph_emlrtRTEI{
    14,                  // lineNo
    15,                  // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo qh_emlrtRTEI{
    16,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace b_signal {
namespace internal {
namespace resample {
void uniformResampleAlongFirstDim(
    emxArray_creal32_T *cpu_xIn, boolean_T *xIn_outdatedOnCpu,
    emxArray_creal32_T *gpu_xIn, boolean_T *xIn_outdatedOnGpu, real_T opts_p,
    real_T opts_q, const emxArray_real32_T *cpu_opts_filterWithPadding,
    real_T opts_filterDelay)
{
  emxArray_creal32_T gpu_yOut;
  emxArray_creal32_T gpu_yRow;
  emxArray_creal32_T *cpu_yOut;
  emxArray_creal32_T *cpu_yRow;
  emxArray_creal32_T *yRow;
  emxArray_real32_T gpu_opts_filterWithPadding;
  emxArray_real_T *b_y;
  emxArray_real_T *y;
  boolean_T c_opts_filterWithPadding_outdat;
  boolean_T d_opts_filterWithPadding_outdat;
  boolean_T yRow_outdatedOnCpu;
  boolean_T yRow_outdatedOnGpu;
  nvtxRangePushA("#fcn#uniformResampleAlongFirstDim#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_yRow);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_yOut);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_opts_filterWithPadding);
  yRow_outdatedOnCpu = false;
  yRow_outdatedOnGpu = false;
  c_opts_filterWithPadding_outdat = false;
  d_opts_filterWithPadding_outdat = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_yOut, 2, &oh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&y, 2, &ph_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_yRow, 2, &qh_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_y, 2, &re_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&yRow, 1, &mh_emlrtRTEI, true);
  if ((!(opts_p == 1.0)) || (!(opts_q == 1.0))) {
    if ((cpu_xIn->size[0] == 1) || (cpu_xIn->size[1] == 1)) {
      real_T b;
      int32_T b_i;
      int32_T i;
      int32_T u1;
      if ((cpu_xIn->size[0] == 0) || (cpu_xIn->size[1] == 0)) {
        u1 = 0;
      } else {
        i = cpu_xIn->size[0];
        u1 = cpu_xIn->size[1];
        if (i >= u1) {
          u1 = i;
        }
      }
      b = std::ceil(static_cast<real_T>(u1) * opts_p / opts_q);
      if ((cpu_xIn->size[0] == 1) && (cpu_xIn->size[1] == 1)) {
        nvtxMarkA("#upfirdn#" MW_AT_LINE);
        upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu,
                (emxArray_real32_T *)cpu_opts_filterWithPadding,
                &c_opts_filterWithPadding_outdat, &gpu_opts_filterWithPadding,
                &d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
        if ((cpu_yRow->size[0] == 0) || (cpu_yRow->size[1] == 0)) {
          u1 = 0;
        } else {
          i = cpu_yRow->size[0];
          u1 = cpu_yRow->size[1];
          if (i >= u1) {
            u1 = i;
          }
        }
        b_i = cpu_yRow->size[0] * cpu_yRow->size[1];
        cpu_yRow->size[0] = u1;
        cpu_yRow->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(cpu_yRow, b_i, &lh_emlrtRTEI);
      } else {
        nvtxMarkA("#upfirdn#" MW_AT_LINE);
        upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu,
                (emxArray_real32_T *)cpu_opts_filterWithPadding,
                &c_opts_filterWithPadding_outdat, &gpu_opts_filterWithPadding,
                &d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
      }
      if (std::isnan(b)) {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_y, b_i, &ib_emlrtRTEI);
        b_y->data[0] = rtNaN;
      } else if (b < 1.0) {
        b_y->size[0] = 1;
        b_y->size[1] = 0;
      } else {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_y, b_i, &ib_emlrtRTEI);
        profileLoopStart("uniformResampleAlongFirstDim_loop_2", __LINE__,
                         static_cast<int32_T>(b - 1.0) + 1, "");
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          b_y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
        profileLoopEnd();
      }
      i = static_cast<int32_T>(b);
      b_i = yRow->size[0];
      yRow->size[0] = static_cast<int32_T>(b);
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(yRow, b_i, &mh_emlrtRTEI);
      profileLoopStart("uniformResampleAlongFirstDim_loop_4", __LINE__,
                       (i - 1) + 1, "");
      for (b_i = 0; b_i < i; b_i++) {
        if (yRow_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal32_T(cpu_yRow, &gpu_yRow);
        }
        yRow_outdatedOnCpu = false;
        yRow->data[b_i] =
            cpu_yRow
                ->data[static_cast<int32_T>(opts_filterDelay + b_y->data[b_i]) -
                       1];
      }
      profileLoopEnd();
      i = static_cast<int32_T>(b);
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = static_cast<int32_T>(b);
      cpu_xIn->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(cpu_xIn, b_i, &nh_emlrtRTEI);
      profileLoopStart("uniformResampleAlongFirstDim_loop_5", __LINE__,
                       (i - 1) + 1, "");
      for (u1 = 0; u1 < i; u1++) {
        if (*xIn_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xIn, gpu_xIn);
        }
        cpu_xIn->data[u1] = yRow->data[u1];
        *xIn_outdatedOnCpu = false;
        *xIn_outdatedOnGpu = true;
      }
      profileLoopEnd();
    } else {
      real_T b;
      int32_T b_i;
      int32_T i;
      b = std::ceil(static_cast<real_T>(cpu_xIn->size[0]) * opts_p / opts_q);
      nvtxMarkA("#upfirdn#" MW_AT_LINE);
      upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu,
              (emxArray_real32_T *)cpu_opts_filterWithPadding,
              &c_opts_filterWithPadding_outdat, &gpu_opts_filterWithPadding,
              &d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yOut,
              &yRow_outdatedOnCpu, &gpu_yOut, &yRow_outdatedOnGpu);
      if (std::isnan(b)) {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, b_i, &ib_emlrtRTEI);
        y->data[0] = rtNaN;
      } else if (b < 1.0) {
        y->size[0] = 1;
        y->size[1] = 0;
      } else {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(y, b_i, &ib_emlrtRTEI);
        profileLoopStart("uniformResampleAlongFirstDim_loop_0", __LINE__,
                         static_cast<int32_T>(b - 1.0) + 1, "");
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
        profileLoopEnd();
      }
      i = cpu_yOut->size[1];
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = y->size[1];
      cpu_xIn->size[1] = cpu_yOut->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(cpu_xIn, b_i, &kh_emlrtRTEI);
      profileLoopStart("uniformResampleAlongFirstDim_loop_1", __LINE__,
                       (i - 1) + 1, "");
      for (b_i = 0; b_i < i; b_i++) {
        profileLoopStart("uniformResampleAlongFirstDim_loop_3", __LINE__,
                         (y->size[1] - 1) + 1, "");
        for (int32_T u1{0}; u1 < y->size[1]; u1++) {
          if (*xIn_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xIn, gpu_xIn);
          }
          if (yRow_outdatedOnCpu) {
            nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
            gpuEmxMemcpyGpuToCpu_creal32_T(cpu_yOut, &gpu_yOut);
          }
          yRow_outdatedOnCpu = false;
          cpu_xIn->data[u1 + cpu_xIn->size[0] * b_i] =
              cpu_yOut
                  ->data[(static_cast<int32_T>(opts_filterDelay + y->data[u1]) +
                          cpu_yOut->size[0] * b_i) -
                         1];
          *xIn_outdatedOnCpu = false;
          *xIn_outdatedOnGpu = true;
        }
        profileLoopEnd();
      }
      profileLoopEnd();
    }
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&yRow);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_y);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_yRow);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&y);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_yOut);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_opts_filterWithPadding);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_yOut);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_yRow);
  nvtxRangePop();
}

} // namespace resample
} // namespace internal
} // namespace b_signal
} // namespace coder

// End of code generation (uniformResampleKernel.cu)

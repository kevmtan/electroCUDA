//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp_initialize.cu
//
// Code generation for function 'ec_wt_fp_initialize'
//

// Include files
#include "ec_wt_fp_initialize.h"
#include "_coder_ec_wt_fp_mex.h"
#include "ec_wt_fp_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void ec_wt_fp_once();

// Function Definitions
static void ec_wt_fp_once()
{
  mex_InitInfAndNan();
  nvtxRangePushA("#initFcn#ec_wt_fp_once#" MW_AT_LOCATION);
  // Initialize GPU by calling hipFree(nullptr)
  nvtxMarkA("#hipFree#" MW_AT_LINE);
  hipFree(nullptr);
  nvtxRangePop();
}

void ec_wt_fp_initialize()
{
  nvtxRangePushA("#fcn#ec_wt_fp_initialize#" MW_AT_LOCATION);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "wavelet_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "signal_toolbox", 2);
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    nvtxMarkA("#ec_wt_fp_once#" MW_AT_LINE);
    ec_wt_fp_once();
  }
  nvtxRangePop();
}

// End of code generation (ec_wt_fp_initialize.cu)

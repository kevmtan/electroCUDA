//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp_terminate.cu
//
// Code generation for function 'ec_wt_fp_terminate'
//

// Include files
#include "ec_wt_fp_terminate.h"
#include "_coder_ec_wt_fp_mex.h"
#include "cwt.h"
#include "ec_wt_fp_data.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void cwt_emx_free_dtor_fcn(const void *r);

static void emlrtExitTimeCleanupDtorFcn(const void *r);

// Function Definitions
static void cwt_emx_free_dtor_fcn(const void *r)
{
  nvtxRangePushA("#fcn#cwt_emx_free_dtor_fcn#" MW_AT_LOCATION);
  nvtxMarkA("#cwt_emx_free#" MW_AT_LINE);
  cwt_emx_free();
  nvtxRangePop();
}

static void emlrtExitTimeCleanupDtorFcn(const void *r)
{
  nvtxRangePushA("#fcn#emlrtExitTimeCleanupDtorFcn#" MW_AT_LOCATION);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  nvtxRangePop();
}

void ec_wt_fp_atexit()
{
  nvtxRangePushA("#termFcn#ec_wt_fp_atexit#" MW_AT_LOCATION);
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  try {
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&emlrtExitTimeCleanupDtorFcn,
                                      nullptr, nullptr, nullptr);
    emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&cwt_emx_free_dtor_fcn, nullptr,
                                      nullptr, nullptr);
    nvtxMarkA("#cwt_emx_free#" MW_AT_LINE);
    cwt_emx_free();
    emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
    emlrtExitTimeCleanup(&emlrtContextGlobal);
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  nvtxRangePop();
}

void ec_wt_fp_terminate()
{
  hipError_t errCode;
  nvtxRangePushA("#fcn#ec_wt_fp_terminate#" MW_AT_LOCATION);
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  nvtxRangePop();
}

// End of code generation (ec_wt_fp_terminate.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwt.cu
//
// Code generation for function 'cwt'
//

// Include files
#include "cwt.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "log2.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hipfft/hipfft.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emxArray_real32_T *psidft;

static boolean_T psidft_not_empty;

static emxArray_real32_T *cf;

static boolean_T cf_not_empty;

static emlrtMCInfo emlrtMCI{
    53,                                                                // lineNo
    14,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtMCInfo b_emlrtMCI{
    55,                                                                // lineNo
    15,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    32,                                                  // lineNo
    12,                                                  // colNo
    "cwt",                                               // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/eml/cwt.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    33,                                                  // lineNo
    12,                                                  // colNo
    "cwt",                                               // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/eml/cwt.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    1,    // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "wt.p" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    1,        // lineNo
    1,        // colNo
    "realWT", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "realWT.p" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    1,             // lineNo
    1,             // colNo
    "cwtFreqGrid", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "cwtFreqGrid.p" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "cuFFTNDCallback", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "cuFFTNDCallback.p" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    1,           // lineNo
    1,           // colNo
    "cwtScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "cwtScales.p" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    1,              // lineNo
    1,              // colNo
    "freqToScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "freqToScales.p" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    1,                // lineNo
    1,                // colNo
    "morseBPFilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "morseBPFilters.p" // pName
};

static emlrtRTEInfo bb_emlrtRTEI{
    104,                                                 // lineNo
    5,                                                   // colNo
    "cwt",                                               // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/eml/cwt.m" // pName
};

static emlrtRTEInfo cb_emlrtRTEI{
    107,                                                 // lineNo
    9,                                                   // colNo
    "cwt",                                               // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/eml/cwt.m" // pName
};

static emlrtRTEInfo db_emlrtRTEI{
    1,                                                   // lineNo
    24,                                                  // colNo
    "cwt",                                               // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/eml/cwt.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret);

static
#ifdef __HIPCC__
    __device__
#endif
        int64_T
        computeEndIdx_device(int64_T start, int64_T end, int64_T stride);

static __global__ void cwt_kernel14(boolean_T b[2], real_T FreqLimits[2],
                                    real_T varargin_2_dim0,
                                    real_T varargin_2_dim1);

static __global__ void cwt_kernel15(const real_T maxscale,
                                    real_T FreqLimits[2]);

static __global__ void cwt_kernel16(const real_T fs, real_T FreqLimits[2]);

static __global__ void cwt_kernel17(const int32_T N, const real_T maxscale,
                                    const int32_T M, emxArray_real_T omega);

static __global__ void cwt_kernel18(const real_T a0, const real_T s0,
                                    const int32_T b, emxArray_real_T scales);

static __global__ void
cwt_kernel19(const real_T fs, const emxArray_real_T omega,
             const emxArray_real_T scales, const int32_T M, const int32_T b,
             const int32_T c, emxArray_real32_T b_cf,
             emxArray_real32_T b_psidft, int32_T psidft_dim0);

static __global__ void cwt_kernel20(const emxArray_real32_T x, const int32_T M,
                                    const int32_T Npad, const int32_T b,
                                    emxArray_real32_T xv);

static __global__ void cwt_kernel21(const emxArray_creal32_T xdft,
                                    const int32_T b, const int32_T c,
                                    emxArray_real32_T b_psidft,
                                    emxArray_creal32_T cfsdft,
                                    int32_T cfsdft_dim0, int32_T psidft_dim0);

static __global__ void cwt_kernel22(const int32_T xSize,
                                    emxArray_creal32_T cfstmp);

static __global__ void cwt_kernel23(const emxArray_creal32_T cfstmp,
                                    const int32_T Npad, const int32_T M,
                                    const int32_T Ns, emxArray_creal32_T cfs,
                                    int32_T cfs_dim0, int32_T cfstmp_dim0);

static __global__ void cwt_kernel24(const emxArray_creal32_T cfstmp,
                                    const int32_T b, const int32_T Ns,
                                    emxArray_creal32_T cfs, int32_T cfs_dim0,
                                    int32_T cfstmp_dim0);

static __global__ void cwt_kernel25(const real32_T cfsdft_re,
                                    const emxArray_creal32_T cfsdft,
                                    const int32_T b_cfsdft,
                                    emxArray_creal32_T cfstmp);

static __global__ void cwt_kernel26(const emxArray_real32_T xv,
                                    const int32_T b_xv,
                                    emxArray_creal32_T xdft);

static __global__ void cwt_kernel27(const int32_T b, emxArray_creal32_T xdft,
                                    uint32_T xSize_dim1);

static __global__ void cwt_kernel28(const emxArray_real32_T x, const int32_T b,
                                    emxArray_real32_T xv);

static __global__ void cwt_kernel29(const real_T a0, const int32_T b,
                                    emxArray_real_T scales);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y);

static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y);

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location);

static real_T rt_powd_snf(real_T u0, real_T u1);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret)
{
  static const int32_T dims[2]{1, -1};
  int32_T iv[2];
  int32_T i;
  boolean_T bv[2]{false, true};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                            (const void *)&dims[0], &bv[0], &iv[0]);
  i = ret->size[0] * ret->size[1];
  ret->size[0] = iv[0];
  ret->size[1] = iv[1];
  nvtxMarkA("#emxEnsureCapacity_char_T#" MW_AT_LINE);
  emxEnsureCapacity_char_T(ret, i, static_cast<emlrtRTEInfo *>(nullptr));
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret->data[0], 1, false);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

static __device__ int64_T computeEndIdx_device(int64_T start, int64_T end,
                                               int64_T stride)
{
  int64_T newEnd;
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  return newEnd;
}

static __global__ __launch_bounds__(32,
                                    1) void cwt_kernel14(boolean_T b[2],
                                                         real_T FreqLimits[2],
                                                         real_T varargin_2_dim0,
                                                         real_T varargin_2_dim1)
{
  __shared__ real_T varargin_2_shared[2];
  int32_T jj;
  if (mwGetThreadIndexWithinBlock() == 0U) {
    varargin_2_shared[0] = varargin_2_dim0;
    varargin_2_shared[1] = varargin_2_dim1;
  }
  __syncthreads();
  jj = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (jj < 2) {
    FreqLimits[jj] = varargin_2_shared[jj];
    b[jj] = isnan(varargin_2_shared[jj]);
  }
}

static __global__ __launch_bounds__(32,
                                    1) void cwt_kernel15(const real_T maxscale,
                                                         real_T FreqLimits[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    FreqLimits[0] = maxscale;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void cwt_kernel16(const real_T fs,
                                                         real_T FreqLimits[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    FreqLimits[1] = fs / 2.0;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void cwt_kernel17(const int32_T N,
                                                         const real_T maxscale,
                                                         const int32_T M,
                                                         emxArray_real_T omega)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(M);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    omega.data[kk] = static_cast<real_T>(kk) * maxscale;
    i = kk + M;
    if (i > 2147483645) {
      i = MAX_int32_T;
    } else {
      i += 2;
    }
    if (i <= N) {
      i = kk + M;
      if (i > 2147483645) {
        i = MAX_int32_T;
      } else {
        i += 2;
      }
      omega.data[i - 1] = 0.0;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel18(
    const real_T a0, const real_T s0, const int32_T b, emxArray_real_T scales)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    scales.data[kk] = s0 * pow(a0, (static_cast<real_T>(kk) + 1.0) - 1.0);
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel19(
    const real_T fs, const emxArray_real_T omega, const emxArray_real_T scales,
    const int32_T M, const int32_T b, const int32_T c, emxArray_real32_T b_cf,
    emxArray_real32_T b_psidft, int32_T psidft_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T jj;
    int32_T kk;
    jj = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    kk = static_cast<int32_T>((idx - static_cast<uint64_T>(jj)) /
                              (static_cast<uint64_T>(b) + 1UL));
    if (jj + 1 > M) {
      b_psidft.data[kk + psidft_dim0 * jj] = 0.0F;
    } else {
      real_T d;
      real_T d1;
      d = scales.data[kk];
      d1 = omega.data[jj];
      b_psidft.data[kk + psidft_dim0 * jj] = static_cast<real32_T>(
          0.0050536085896138571 * exp(20.0 * log(d * d1) - pow(d * d1, 3.0)));
    }
    if (jj + 1 == 1) {
      b_cf.data[kk] = static_cast<real32_T>(
          1.8820720577620569 / scales.data[kk] / 6.2831853071795862 * fs);
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel20(
    const emxArray_real32_T x, const int32_T M, const int32_T Npad,
    const int32_T b, emxArray_real32_T xv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    if (kk < Npad) {
      xv.data[kk] = x.data[(Npad - kk) - 1];
    } else {
      int32_T qY;
      boolean_T guard1;
      guard1 = false;
      if (kk + 1 > Npad) {
        if (M > MAX_int32_T - Npad) {
          qY = MAX_int32_T;
        } else {
          qY = Npad + M;
        }
        if (kk + 1 <= qY) {
          qY = (Npad - kk) - 1;
          if (qY < 0) {
            qY = -qY;
          }
          xv.data[kk] = x.data[qY - 1];
        } else {
          guard1 = true;
        }
      } else {
        guard1 = true;
      }
      if (guard1) {
        if ((M > 0) && (Npad > MAX_int32_T - M)) {
          qY = MAX_int32_T;
        } else {
          qY = M + Npad;
        }
        if (kk + 1 > qY) {
          if (M > 1073741823) {
            qY = MAX_int32_T;
          } else {
            qY = M << 1;
          }
          qY = (qY - kk) - 1;
          if ((qY < 0) && (Npad < MIN_int32_T - qY)) {
            qY = MIN_int32_T;
          } else if ((qY > 0) && (Npad > MAX_int32_T - qY)) {
            qY = MAX_int32_T;
          } else {
            qY += Npad;
          }
          if (qY <= 2147483646) {
            qY++;
          }
          xv.data[kk] = x.data[qY - 1];
        }
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel21(
    const emxArray_creal32_T xdft, const int32_T b, const int32_T c,
    emxArray_real32_T b_psidft, emxArray_creal32_T cfsdft, int32_T cfsdft_dim0,
    int32_T psidft_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T jj;
    int32_T kk;
    jj = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    kk = static_cast<int32_T>((idx - static_cast<uint64_T>(jj)) /
                              (static_cast<uint64_T>(b) + 1UL));
    cfsdft.data[kk + cfsdft_dim0 * jj].re =
        b_psidft.data[kk + psidft_dim0 * jj] * xdft.data[jj].re;
    cfsdft.data[kk + cfsdft_dim0 * jj].im =
        b_psidft.data[kk + psidft_dim0 * jj] * xdft.data[jj].im;
  }
}

static __global__
    __launch_bounds__(1024, 1) void cwt_kernel22(const int32_T xSize,
                                                 emxArray_creal32_T cfstmp)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(xSize);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    cfstmp.data[jj].re = 0.0F;
    cfstmp.data[jj].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel23(
    const emxArray_creal32_T cfstmp, const int32_T Npad, const int32_T M,
    const int32_T Ns, emxArray_creal32_T cfs, int32_T cfs_dim0,
    int32_T cfstmp_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(Ns);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T jj;
    int32_T qY;
    i = static_cast<int32_T>(idx);
    if (M > MAX_int32_T - Npad) {
      qY = MAX_int32_T;
    } else {
      qY = Npad + M;
    }
    jj = Npad + 1;
    for (int64_T b_jj{0L};
         b_jj <= computeEndIdx_device(static_cast<int64_T>(jj),
                                      static_cast<int64_T>(qY), 1L);
         b_jj++) {
      cfs.data[i + cfs_dim0 * ((static_cast<int32_T>(
                                    static_cast<int64_T>(Npad + 1) + b_jj) -
                                Npad) -
                               1)] =
          cfstmp.data[i + cfstmp_dim0 *
                              (static_cast<int32_T>(
                                   static_cast<int64_T>(Npad + 1) + b_jj) -
                               1)];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel24(
    const emxArray_creal32_T cfstmp, const int32_T b, const int32_T Ns,
    emxArray_creal32_T cfs, int32_T cfs_dim0, int32_T cfstmp_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(Ns) + 1UL) * (static_cast<uint64_T>(b) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T jj;
    jj = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(jj)) /
                             (static_cast<uint64_T>(b) + 1UL));
    cfs.data[i + cfs_dim0 * jj] = cfstmp.data[i + cfstmp_dim0 * jj];
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel25(
    const real32_T cfsdft_re, const emxArray_creal32_T cfsdft,
    const int32_T b_cfsdft, emxArray_creal32_T cfstmp)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_cfsdft);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T jj;
    real32_T ai;
    real32_T ar;
    jj = static_cast<int32_T>(idx);
    ar = cfsdft.data[jj].re;
    ai = cfsdft.data[jj].im;
    if (ai == 0.0F) {
      cfstmp.data[jj].re = ar / cfsdft_re;
      cfstmp.data[jj].im = 0.0F;
    } else if (ar == 0.0F) {
      cfstmp.data[jj].re = 0.0F;
      cfstmp.data[jj].im = ai / cfsdft_re;
    } else {
      cfstmp.data[jj].re = ar / cfsdft_re;
      cfstmp.data[jj].im = ai / cfsdft_re;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel26(
    const emxArray_real32_T xv, const int32_T b_xv, emxArray_creal32_T xdft)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_xv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    xdft.data[jj].re = xv.data[jj];
    xdft.data[jj].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel27(
    const int32_T b, emxArray_creal32_T xdft, uint32_T xSize_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    xdft.data[(static_cast<int32_T>(xSize_dim1) - i) - 1].re =
        xdft.data[i + 1].re;
    xdft.data[(static_cast<int32_T>(xSize_dim1) - i) - 1].im =
        -xdft.data[i + 1].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void cwt_kernel28(
    const emxArray_real32_T x, const int32_T b, emxArray_real32_T xv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    xv.data[kk] = x.data[kk];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void cwt_kernel29(const real_T a0,
                                                         const int32_T b,
                                                         emxArray_real_T scales)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    scales.data[kk] =
        0.68998891650046823 * pow(a0, (static_cast<real_T>(kk) + 1.0) - 1.0);
  }
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y)
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y)
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(tmpStr), &thisId, y);
  emlrtDestroyArray(&tmpStr);
  nvtxRangePop();
}

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location)
{
  const mxArray *pArrays[3];
  const mxArray *m;
  const mxArray *m4;
  nvtxRangePushA("#fcn#feval#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  pArrays[2] = m3;
  m4 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 3, &pArrays[0], "feval",
                             true, location);
  nvtxRangePop();
  return m4;
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu,
                                        boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real_T));
  nvtxRangePop();
}

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location)
{
  const mxArray *m;
  const mxArray *m2;
  const mxArray *pArray;
  nvtxRangePushA("#fcn#length#" MW_AT_LOCATION);
  pArray = m1;
  m2 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 1, &pArray, "length",
                             true, location);
  nvtxRangePop();
  return m2;
}

static real_T rt_powd_snf(real_T u0, real_T u1)
{
  real_T y;
  nvtxRangePushA("#fcn#rt_powd_snf#" MW_AT_LOCATION);
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else {
    real_T b;
    real_T c;
    b = std::abs(u0);
    c = std::abs(u1);
    if (std::isinf(u1)) {
      if (b == 1.0) {
        y = 1.0;
      } else if (b > 1.0) {
        if (u1 > 0.0) {
          y = rtInf;
        } else {
          y = 0.0;
        }
      } else if (u1 > 0.0) {
        y = 0.0;
      } else {
        y = rtInf;
      }
    } else if (c == 0.0) {
      y = 1.0;
    } else if (c == 1.0) {
      if (u1 > 0.0) {
        y = u0;
      } else {
        y = 1.0 / u0;
      }
    } else if (u1 == 2.0) {
      y = u0 * u0;
    } else if ((u1 == 0.5) && (u0 >= 0.0)) {
      y = std::sqrt(u0);
    } else if ((u0 < 0.0) && (u1 > std::floor(u1))) {
      y = rtNaN;
    } else {
      y = std::pow(u0, u1);
    }
  }
  nvtxRangePop();
  return y;
}

//
//
namespace coder {
void cwt(emxArray_real32_T *cpu_x, emxArray_real32_T *gpu_x,
         boolean_T *x_outdatedOnGpu, real_T fs, const real_T varargin_2[2],
         real_T varargin_4, emxArray_creal32_T *cpu_cfs,
         boolean_T *cfs_outdatedOnCpu, emxArray_creal32_T *gpu_cfs,
         boolean_T *cfs_outdatedOnGpu)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 5};
  static const char_T u[7]{'s', 'p', 'r', 'i', 'n', 't', 'f'};
  static const char_T formatSpec[5]{'%', '2', '.', '2', 'f'};
  dim3 block;
  dim3 grid;
  emxArray_char_T *b_tmpStr;
  emxArray_creal32_T gpu_cfsdft;
  emxArray_creal32_T gpu_cfstmp;
  emxArray_creal32_T gpu_xdft;
  emxArray_creal32_T *cpu_cfsdft;
  emxArray_creal32_T *cpu_cfstmp;
  emxArray_creal32_T *cpu_xdft;
  emxArray_real32_T gpu_cf;
  emxArray_real32_T gpu_psidft;
  emxArray_real32_T gpu_xv;
  emxArray_real32_T *cpu_xv;
  emxArray_real_T gpu_omega;
  emxArray_real_T gpu_scales;
  emxArray_real_T *cpu_omega;
  emxArray_real_T *cpu_scales;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *tmpStr;
  real_T(*gpu_FreqLimits)[2];
  int32_T M;
  int32_T N;
  int32_T Nomega;
  int32_T Npad;
  int32_T Ns;
  int32_T i;
  uint32_T xSize[2];
  boolean_T(*gpu_b)[2];
  boolean_T validLaunchParams;
  boolean_T xdft_needsGpuEnsureCapacity;
  nvtxRangePushA("#fcn#cwt#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_cf);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_psidft);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_scales);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_cfsdft);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_omega);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_xdft);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_cfstmp);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_xv);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_b, 2UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_FreqLimits, 16UL), __FILE__, __LINE__);
  xdft_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if (!psidft_not_empty) {
    real_T cpu_FreqLimits[2];
    real_T maxscale;
    boolean_T cpu_b[2];
    boolean_T FreqLimits_outdatedOnCpu;
    boolean_T b_outdatedOnCpu;
    boolean_T exitg1;
    boolean_T y;
    N = static_cast<int32_T>(
        std::trunc(static_cast<real_T>(cpu_x->size[0]) / 2.0));
    nvtxMarkA("#cwt_kernel14#" MW_AT_LINE);
    cwt_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_b, *gpu_FreqLimits, varargin_2[0], varargin_2[1]);
    FreqLimits_outdatedOnCpu = true;
    b_outdatedOnCpu = true;
    y = true;
    Nomega = 0;
    exitg1 = false;
    nvtxRangePushA("#loop#cwt_whileloop_1##" MW_AT_LINE);
    while ((!exitg1) && (Nomega < 2)) {
      if (b_outdatedOnCpu) {
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(hipMemcpy(cpu_b, *gpu_b, 2UL, hipMemcpyDeviceToHost),
                       __FILE__, __LINE__);
      }
      b_outdatedOnCpu = false;
      if (!cpu_b[Nomega]) {
        y = false;
        exitg1 = true;
      } else {
        Nomega++;
      }
    }
    nvtxRangePop();
    if (!y) {
      maxscale = static_cast<real_T>(cpu_x->size[0]) / 11.001547073884861;
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      if (maxscale < 0.68998891650046823 * rt_powd_snf(2.0, 1.0 / varargin_4)) {
        nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
        maxscale = 0.68998891650046823 * rt_powd_snf(2.0, 1.0 / varargin_4);
      }
      maxscale = 1.0 / (maxscale * 3.3384403542183319) * fs;
      if (varargin_2[0] < maxscale) {
        nvtxMarkA("#cwt_kernel15#" MW_AT_LINE);
        cwt_kernel15<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(maxscale,
                                                              *gpu_FreqLimits);
      }
      if (varargin_2[1] > fs / 2.0) {
        nvtxMarkA("#cwt_kernel16#" MW_AT_LINE);
        cwt_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(fs,
                                                              *gpu_FreqLimits);
      }
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(hipMemcpy(cpu_FreqLimits, *gpu_FreqLimits, 16UL,
                                hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
      FreqLimits_outdatedOnCpu = false;
      nvtxMarkA("#b_log2#" MW_AT_LINE);
      nvtxMarkA("#b_log2#" MW_AT_LINE);
      if (!(b_log2(cpu_FreqLimits[1]) - b_log2(cpu_FreqLimits[0]) >=
            1.0 / varargin_4)) {
        tmpStr = nullptr;
        b_y = nullptr;
        m = emlrtCreateCharArray(2, &iv[0]);
        emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &u[0]);
        emlrtAssign(&b_y, m);
        c_y = nullptr;
        m1 = emlrtCreateCharArray(2, &iv1[0]);
        emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m1, &formatSpec[0]);
        emlrtAssign(&c_y, m1);
        d_y = nullptr;
        m2 = emlrtCreateDoubleScalar(1.0 / varargin_4);
        emlrtAssign(&d_y, m2);
        nvtxMarkA("#feval#" MW_AT_LINE);
        emlrtAssign(&tmpStr, feval(b_y, c_y, d_y, &emlrtMCI));
        nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
        nvtxMarkA("#length#" MW_AT_LINE);
        emlrt_marshallIn(length(emlrtAlias(tmpStr), &b_emlrtMCI),
                         "<output of length>");
        nvtxMarkA("#emxInit_char_T#" MW_AT_LINE);
        emxInit_char_T(&b_tmpStr, 2, &db_emlrtRTEI, true);
        nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
        emlrt_marshallIn(emlrtAlias(tmpStr), "tmpStr", b_tmpStr);
        nvtxMarkA("#emxFree_char_T#" MW_AT_LINE);
        emxFree_char_T(&b_tmpStr);
        emlrtDestroyArray(&tmpStr);
      }
    }
    if (cpu_x->size[0] > 100000) {
      nvtxMarkA("#b_log2#" MW_AT_LINE);
      maxscale = std::ceil(b_log2(static_cast<real_T>(cpu_x->size[0])));
      if (maxscale < 2.147483648E+9) {
        if (maxscale >= -2.147483648E+9) {
          N = static_cast<int32_T>(maxscale);
        } else {
          N = MIN_int32_T;
        }
      } else if (maxscale >= 2.147483648E+9) {
        N = MAX_int32_T;
      } else {
        N = 0;
      }
    }
    if (N > 1073741823) {
      Nomega = MAX_int32_T;
    } else if (N <= -1073741824) {
      Nomega = MIN_int32_T;
    } else {
      Nomega = N << 1;
    }
    if ((cpu_x->size[0] > 0) && (Nomega > MAX_int32_T - cpu_x->size[0])) {
      N = MAX_int32_T;
    } else {
      N = cpu_x->size[0] + Nomega;
    }
    M = static_cast<int32_T>(std::trunc(static_cast<real_T>(N) / 2.0));
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_omega, 2, &bb_emlrtRTEI, true);
    i = cpu_omega->size[0] * cpu_omega->size[1];
    cpu_omega->size[0] = 1;
    cpu_omega->size[1] = N;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_omega, i, &v_emlrtRTEI);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(M), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_omega, &gpu_omega, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel17#" MW_AT_LINE);
      cwt_kernel17<<<grid, block>>>(
          N, 6.2831853071795862 / static_cast<real_T>(N), M, gpu_omega);
    }
    profileLoopStart("cwt_loop_1", __LINE__, 1 + 1, "");
    for (Nomega = 0; Nomega < 2; Nomega++) {
      if (FreqLimits_outdatedOnCpu) {
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(hipMemcpy(cpu_FreqLimits, *gpu_FreqLimits, 16UL,
                                  hipMemcpyDeviceToHost),
                       __FILE__, __LINE__);
      }
      FreqLimits_outdatedOnCpu = false;
      cpu_b[Nomega] = std::isnan(cpu_FreqLimits[Nomega]);
    }
    profileLoopEnd();
    y = true;
    Nomega = 0;
    exitg1 = false;
    nvtxRangePushA("#loop#cwt_whileloop_0##" MW_AT_LINE);
    while ((!exitg1) && (Nomega < 2)) {
      if (!cpu_b[Nomega]) {
        y = false;
        exitg1 = true;
      } else {
        Nomega++;
      }
    }
    nvtxRangePop();
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_scales, 2, &cb_emlrtRTEI, true);
    if (!y) {
      real_T s0;
      profileLoopStart("cwt_loop_2", __LINE__, 1 + 1, "");
      for (Nomega = 0; Nomega < 2; Nomega++) {
        if (FreqLimits_outdatedOnCpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(hipMemcpy(cpu_FreqLimits, *gpu_FreqLimits, 16UL,
                                    hipMemcpyDeviceToHost),
                         __FILE__, __LINE__);
        }
        cpu_FreqLimits[Nomega] =
            cpu_FreqLimits[Nomega] / fs * 2.0 * 3.1415926535897931;
        FreqLimits_outdatedOnCpu = false;
      }
      profileLoopEnd();
      if (FreqLimits_outdatedOnCpu) {
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(hipMemcpy(cpu_FreqLimits, *gpu_FreqLimits, 16UL,
                                  hipMemcpyDeviceToHost),
                       __FILE__, __LINE__);
      }
      s0 = 1.8820720577620569 / cpu_FreqLimits[1];
      nvtxMarkA("#b_log2#" MW_AT_LINE);
      maxscale = std::floor(
          varargin_4 * b_log2(1.8820720577620569 / cpu_FreqLimits[0] / s0));
      if (maxscale < 2.147483648E+9) {
        if (maxscale >= -2.147483648E+9) {
          Nomega = static_cast<int32_T>(maxscale);
        } else {
          Nomega = MIN_int32_T;
        }
      } else if (maxscale >= 2.147483648E+9) {
        Nomega = MAX_int32_T;
      } else {
        Nomega = 0;
      }
      if (Nomega > 2147483646) {
        Ns = MAX_int32_T;
      } else {
        Ns = Nomega + 1;
      }
      i = cpu_scales->size[0] * cpu_scales->size[1];
      cpu_scales->size[0] = 1;
      cpu_scales->size[1] = Ns;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_scales, i, &y_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(Ns - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_scales, &gpu_scales, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#cwt_kernel18#" MW_AT_LINE);
        nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
        cwt_kernel18<<<grid, block>>>(rt_powd_snf(2.0, 1.0 / varargin_4), s0,
                                      Ns - 1, gpu_scales);
      }
    } else {
      maxscale = static_cast<real_T>(cpu_x->size[0]) / 11.001547073884861;
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      if (maxscale < 0.68998891650046823 * rt_powd_snf(2.0, 1.0 / varargin_4)) {
        nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
        maxscale = 0.68998891650046823 * rt_powd_snf(2.0, 1.0 / varargin_4);
      }
      nvtxMarkA("#b_log2#" MW_AT_LINE);
      maxscale = std::floor(
          std::fmax(b_log2(maxscale / 0.68998891650046823), 1.0 / varargin_4) *
          varargin_4);
      if (maxscale < 2.147483648E+9) {
        if (maxscale >= -2.147483648E+9) {
          Nomega = static_cast<int32_T>(maxscale);
        } else {
          Nomega = MIN_int32_T;
        }
      } else if (maxscale >= 2.147483648E+9) {
        Nomega = MAX_int32_T;
      } else {
        Nomega = 0;
      }
      if (Nomega > 2147483646) {
        Ns = MAX_int32_T;
      } else {
        Ns = Nomega + 1;
      }
      i = cpu_scales->size[0] * cpu_scales->size[1];
      cpu_scales->size[0] = 1;
      cpu_scales->size[1] = Ns;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_scales, i, &x_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(Ns - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_scales, &gpu_scales, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#cwt_kernel29#" MW_AT_LINE);
        nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
        cwt_kernel29<<<grid, block>>>(rt_powd_snf(2.0, 1.0 / varargin_4),
                                      Ns - 1, gpu_scales);
      }
    }
    Ns = cpu_scales->size[1];
    Nomega = cpu_omega->size[1];
    i = psidft->size[0] * psidft->size[1];
    psidft->size[0] = cpu_scales->size[1];
    psidft->size[1] = cpu_omega->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(psidft, i, &ab_emlrtRTEI);
    i = cf->size[0];
    cf->size[0] = cpu_scales->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cf, i, &ab_emlrtRTEI);
    M = static_cast<int32_T>(
            std::trunc(static_cast<real_T>(cpu_omega->size[1]) / 2.0)) +
        1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(Ns - 1, Nomega - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cf, &gpu_cf, false);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(psidft, &gpu_psidft, false);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_cf, cf);
    nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_psidft, psidft);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel19#" MW_AT_LINE);
      cwt_kernel19<<<grid, block>>>(fs, gpu_omega, gpu_scales, M, Nomega - 1,
                                    Ns - 1, gpu_cf, gpu_psidft,
                                    psidft->size[0U]);
    }
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real32_T(psidft, &gpu_psidft);
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real32_T(cf, &gpu_cf);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_scales);
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_omega);
    psidft_not_empty = ((psidft->size[0] != 0) && (psidft->size[1] != 0));
    cf_not_empty = (cf->size[0] != 0);
  }
  M = cpu_x->size[0];
  Npad = static_cast<int32_T>(
      std::trunc(static_cast<real_T>(psidft->size[1] - cpu_x->size[0]) / 2.0));
  Ns = psidft->size[0] - 1;
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_xv, 2, &t_emlrtRTEI, true);
  i = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = psidft->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(cpu_xv, i, &t_emlrtRTEI);
  i = cpu_cfs->size[0] * cpu_cfs->size[1];
  cpu_cfs->size[0] = psidft->size[0];
  cpu_cfs->size[1] = cpu_x->size[0];
  nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
  emxEnsureCapacity_creal32_T(cpu_cfs, i, &t_emlrtRTEI);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_cfstmp, 2, &t_emlrtRTEI, true);
  N = psidft->size[1];
  if (Npad > 0) {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(N - 1), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, true);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel20#" MW_AT_LINE);
      cwt_kernel20<<<grid, block>>>(*gpu_x, M, Npad, N - 1, gpu_xv);
    }
  } else {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(M - 1), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, true);
    if (*x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel28#" MW_AT_LINE);
      cwt_kernel28<<<grid, block>>>(*gpu_x, M - 1, gpu_xv);
    }
  }
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_xdft, 2, &u_emlrtRTEI, true);
  if (cpu_xv->size[1] == 0) {
    cpu_xdft->size[0] = 1;
    cpu_xdft->size[1] = 0;
  } else if (cpu_xv->size[1] - (static_cast<int32_T>(
                                    static_cast<uint32_T>(cpu_xv->size[1]) >> 1)
                                << 1) ==
             1) {
    hipfftHandle fftPlanHandle;
    i = cpu_xdft->size[0] * cpu_xdft->size[1];
    cpu_xdft->size[0] = 1;
    cpu_xdft->size[1] = cpu_xv->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_xdft, i, &u_emlrtRTEI);
    Nomega = cpu_xv->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(Nomega), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_xdft, &gpu_xdft, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel26#" MW_AT_LINE);
      cwt_kernel26<<<grid, block>>>(gpu_xv, Nomega, gpu_xdft);
    }
    N = cpu_xdft->size[1];
    fftPlanHandle = acquireCUFFTPlan(1, &N, &N, 1, 1, HIPFFT_C2C, 1);
    xdft_needsGpuEnsureCapacity = false;
    hipfftExecC2C(fftPlanHandle, (hipfftComplex *)&gpu_xdft.data[0],
                 (hipfftComplex *)&gpu_xdft.data[0], HIPFFT_FORWARD);
  } else {
    N = cpu_xv->size[1];
    profileLoopStart("cwt_loop_0", __LINE__, 1 + 1, "");
    for (Nomega = 0; Nomega < 2; Nomega++) {
      xSize[Nomega] = static_cast<uint32_T>(cpu_xv->size[Nomega]);
    }
    hipfftHandle b_fftPlanHandle;
    profileLoopEnd();
    i = cpu_xdft->size[0] * cpu_xdft->size[1];
    cpu_xdft->size[0] = 1;
    cpu_xdft->size[1] = static_cast<int32_T>(xSize[1]);
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_xdft, i, &w_emlrtRTEI);
    b_fftPlanHandle = acquireCUFFTPlan(1, &N, &N, 1, 1, HIPFFT_R2C, 1);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_xdft, &gpu_xdft, true);
    hipfftExecR2C(b_fftPlanHandle, (hipfftReal *)&gpu_xv.data[0],
                 (hipfftComplex *)&gpu_xdft.data[0]);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(
        computeNumIters(
            static_cast<int32_T>(
                static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) / 2.0) -
            2),
        &grid, &block, 2147483647U);
    xdft_needsGpuEnsureCapacity = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel27#" MW_AT_LINE);
      cwt_kernel27<<<grid, block>>>(
          static_cast<int32_T>(
              static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) / 2.0) -
              2,
          gpu_xdft, xSize[1]);
    }
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_xv);
  N = psidft->size[0];
  Nomega = psidft->size[1];
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_cfsdft, 2, &u_emlrtRTEI, true);
  i = cpu_cfsdft->size[0] * cpu_cfsdft->size[1];
  cpu_cfsdft->size[0] = psidft->size[0];
  cpu_cfsdft->size[1] = psidft->size[1];
  nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
  emxEnsureCapacity_creal32_T(cpu_cfsdft, i, &u_emlrtRTEI);
  nvtxMarkA("#computeNumIters#" MW_AT_LINE);
  mwGetLaunchParameters1D(computeNumIters(N - 1, Nomega - 1), &grid, &block,
                          2147483647U);
  if (xdft_needsGpuEnsureCapacity) {
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_xdft, &gpu_xdft, true);
  }
  nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_real32_T(psidft, &gpu_psidft, false);
  nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
  gpuEmxEnsureCapacity_creal32_T(cpu_cfsdft, &gpu_cfsdft, true);
  nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
  gpuEmxMemcpyCpuToGpu_real32_T(&gpu_psidft, psidft);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    nvtxMarkA("#cwt_kernel21#" MW_AT_LINE);
    cwt_kernel21<<<grid, block>>>(gpu_xdft, Nomega - 1, N - 1, gpu_psidft,
                                  gpu_cfsdft, cpu_cfsdft->size[0U],
                                  psidft->size[0U]);
  }
  nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
  gpuEmxMemcpyGpuToCpu_real32_T(psidft, &gpu_psidft);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_xdft);
  if ((cpu_cfsdft->size[0] == 0) || (cpu_cfsdft->size[1] == 0)) {
    profileLoopStart("cwt_loop_3", __LINE__, 1 + 1, "");
    for (Nomega = 0; Nomega < 2; Nomega++) {
      xSize[Nomega] = static_cast<uint32_T>(cpu_cfsdft->size[Nomega]);
    }
    profileLoopEnd();
    xSize[1] = static_cast<uint32_T>(cpu_cfsdft->size[1]);
    i = cpu_cfstmp->size[0] * cpu_cfstmp->size[1];
    cpu_cfstmp->size[0] = static_cast<int32_T>(xSize[0]);
    cpu_cfstmp->size[1] = cpu_cfsdft->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_cfstmp, i, &t_emlrtRTEI);
    Nomega =
        static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(Nomega), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_cfstmp, &gpu_cfstmp, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel22#" MW_AT_LINE);
      cwt_kernel22<<<grid, block>>>(Nomega, gpu_cfstmp);
    }
  } else {
    hipfftHandle c_fftPlanHandle;
    real32_T cfsdft_re;
    N = cpu_cfsdft->size[1];
    c_fftPlanHandle = acquireCUFFTPlan(1, &N, &N, cpu_cfsdft->size[0], 1,
                                       HIPFFT_C2C, cpu_cfsdft->size[0]);
    hipfftExecC2C(c_fftPlanHandle, (hipfftComplex *)&gpu_cfsdft.data[0],
                 (hipfftComplex *)&gpu_cfsdft.data[0], HIPFFT_BACKWARD);
    i = cpu_cfstmp->size[0] * cpu_cfstmp->size[1];
    cpu_cfstmp->size[0] = cpu_cfsdft->size[0];
    cpu_cfstmp->size[1] = cpu_cfsdft->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_cfstmp, i, &t_emlrtRTEI);
    cfsdft_re = static_cast<real32_T>(cpu_cfsdft->size[1]);
    Nomega = cpu_cfsdft->size[0] * cpu_cfsdft->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(Nomega), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_cfstmp, &gpu_cfstmp, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel25#" MW_AT_LINE);
      cwt_kernel25<<<grid, block>>>(cfsdft_re, gpu_cfsdft, Nomega, gpu_cfstmp);
    }
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_cfsdft);
  if (Npad > 0) {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(Ns), &grid, &block, 2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_cfs, gpu_cfs, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel23#" MW_AT_LINE);
      cwt_kernel23<<<grid, block>>>(gpu_cfstmp, Npad, M, Ns, *gpu_cfs,
                                    cpu_cfs->size[0U], cpu_cfstmp->size[0U]);
    }
  } else {
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(Ns, M - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_cfs, gpu_cfs, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwt_kernel24#" MW_AT_LINE);
      cwt_kernel24<<<grid, block>>>(gpu_cfstmp, M - 1, Ns, *gpu_cfs,
                                    cpu_cfs->size[0U], cpu_cfstmp->size[0U]);
    }
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_cfstmp);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_FreqLimits), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_b), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_xv);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_cfstmp);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_xdft);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_omega);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_cfsdft);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_scales);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_psidft);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_cf);
  *x_outdatedOnGpu = false;
  *cfs_outdatedOnCpu = true;
  *cfs_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace coder
void cwt_emx_free()
{
  nvtxRangePushA("#fcn#cwt_emx_free#" MW_AT_LOCATION);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&psidft);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cf);
  nvtxRangePop();
}

void cwt_emx_init()
{
  nvtxRangePushA("#fcn#cwt_emx_init#" MW_AT_LOCATION);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&psidft, 2, &emlrtRTEI, false);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cf, 1, &b_emlrtRTEI, false);
  nvtxRangePop();
}

void cwt_init()
{
  nvtxRangePushA("#fcn#cwt_init#" MW_AT_LOCATION);
  cf_not_empty = false;
  psidft_not_empty = false;
  nvtxRangePop();
}

// End of code generation (cwt.cu)

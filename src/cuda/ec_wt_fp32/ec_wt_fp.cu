#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp.cu
//
// Code generation for function 'ec_wt_fp'
//

// Include files
#include "ec_wt_fp.h"
#include "circshift.h"
#include "cwt.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Variable Definitions
static emlrtMCInfo c_emlrtMCI{
    53,        // lineNo
    19,        // colNo
    "flt2str", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/coder/coder/lib/+coder/+internal/"
    "flt2str.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    24,                                                     // lineNo
    27,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    32,                                                     // lineNo
    25,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    30,                                                     // lineNo
    26,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    43,                                                     // lineNo
    1,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    55,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    32,                                                     // lineNo
    9,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    30,                    // lineNo
    21,                    // colNo
    "applyScalarFunction", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "applyScalarFunction.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    30,                                                     // lineNo
    9,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    91,                                                           // lineNo
    14,                                                           // colNo
    "downsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/downsample.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    129,                         // lineNo
    6,                           // colNo
    "applyBinaryScalarFunction", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "applyBinaryScalarFunction.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    59,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    47,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    64,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    30,                                                     // lineNo
    17,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    32,                                                     // lineNo
    17,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    59,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    43,                                                     // lineNo
    6,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23]);

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location);

static int32_T div_s32(int32_T numerator, int32_T denominator);

static
#ifdef __HIPCC__
    __device__
#endif
        int32_T
        div_s32_device(int32_T numerator, int32_T denominator);

static __global__ void ec_wt_fp_kernel1(const int32_T ch,
                                        const emxArray_real32_T x,
                                        const int32_T b_x,
                                        emxArray_real32_T c_x, int32_T x_dim0);

static __global__ void ec_wt_fp_kernel10(const emxArray_creal32_T r,
                                         const int32_T b_r, const int32_T c_r,
                                         emxArray_creal32_T yc, int32_T yc_dim0,
                                         int32_T r_dim0);

static __global__ void ec_wt_fp_kernel11(const int32_T i1, const int32_T i2,
                                         const int32_T nx, int32_T sizeX[2]);

static __global__ void
ec_wt_fp_kernel12(const emxArray_creal32_T yc, const int32_T lshift_dim0,
                  const int32_T i1, const int32_T b, const int32_T c,
                  emxArray_creal32_T b_yc, int32_T yc_dim0);

static __global__ void ec_wt_fp_kernel13(const emxArray_creal32_T yc,
                                         const int32_T lshift,
                                         emxArray_creal32_T b_yc);

static __global__ void ec_wt_fp_kernel2(const emxArray_creal32_T r1,
                                        const int32_T b_r1, const int32_T c_r1,
                                        emxArray_creal32_T x, int32_T x_dim0,
                                        int32_T r1_dim0);

static __global__ void ec_wt_fp_kernel3(const emxArray_creal32_T x,
                                        const int32_T b, emxArray_real32_T yc);

static __global__ void ec_wt_fp_kernel4(const emxArray_real32_T yc,
                                        const int32_T b, emxArray_real32_T y);

static __global__ void ec_wt_fp_kernel5(const emxArray_real32_T y,
                                        const int32_T b_y,
                                        emxArray_real32_T yc);

static __global__ void ec_wt_fp_kernel6(const int32_T i1, const int32_T i2,
                                        const int32_T nx, int32_T sizeX[2]);

static __global__ void ec_wt_fp_kernel7(const emxArray_real32_T yc,
                                        const int32_T lshift_dim0,
                                        const int32_T i1, const int32_T b,
                                        const int32_T c, emxArray_real32_T b_yc,
                                        int32_T yc_dim0);

static __global__ void ec_wt_fp_kernel8(const emxArray_real32_T yc,
                                        const int32_T lshift,
                                        emxArray_real32_T b_yc);

static __global__ void ec_wt_fp_kernel9(const int32_T ch,
                                        const emxArray_real32_T x,
                                        const int32_T b_x,
                                        emxArray_real32_T c_x, int32_T x_dim0);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23]);

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23]);

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu);

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23])
{
  static const int32_T dims[2]{1, 23};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 23);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  const mxArray *m3;
  nvtxRangePushA("#fcn#b_sprintf#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  m3 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                             "sprintf", true, location);
  nvtxRangePop();
  return m3;
}

static int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  nvtxRangePushA("#fcn#div_s32#" MW_AT_LOCATION);
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  nvtxRangePop();
  return quotient;
}

static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if (static_cast<int32_T>(numerator < 0) !=
        static_cast<int32_T>(denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel1(
    const int32_T ch, const emxArray_real32_T x, const int32_T b_x,
    emxArray_real32_T c_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_x.data[i1] = x.data[i1 + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel10(
    const emxArray_creal32_T r, const int32_T b_r, const int32_T c_r,
    emxArray_creal32_T yc, int32_T yc_dim0, int32_T r_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_r) + 1UL) * (static_cast<uint64_T>(b_r) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_r) + 1UL));
    yc.data[k + yc_dim0 * i1].re = r.data[i1 + r_dim0 * k].re;
    yc.data[k + yc_dim0 * i1].im = -r.data[i1 + r_dim0 * k].im;
  }
}

static __global__ __launch_bounds__(32, 1) void ec_wt_fp_kernel11(
    const int32_T i1, const int32_T i2, const int32_T nx, int32_T sizeX[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    sizeX[nx] = div_s32_device(i2, i1) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel12(
    const emxArray_creal32_T yc, const int32_T lshift_dim0, const int32_T i1,
    const int32_T b, const int32_T c, emxArray_creal32_T b_yc, int32_T yc_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    b_i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_yc.data[k + yc_dim0 * b_i1] = yc.data[i1 * k + lshift_dim0 * b_i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel13(
    const emxArray_creal32_T yc, const int32_T lshift, emxArray_creal32_T b_yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(lshift);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_yc.data[i1] = yc.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel2(
    const emxArray_creal32_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_creal32_T x, int32_T x_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_r1) + 1UL));
    x.data[k + x_dim0 * i1].re = r1.data[i1 + r1_dim0 * k].re;
    x.data[k + x_dim0 * i1].im = -r1.data[i1 + r1_dim0 * k].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel3(
    const emxArray_creal32_T x, const int32_T b, emxArray_real32_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    yc.data[k] = hypotf(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel4(
    const emxArray_real32_T yc, const int32_T b, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = yc.data[k] * yc.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel5(
    const emxArray_real32_T y, const int32_T b_y, emxArray_real32_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    yc.data[i1] = y.data[i1];
  }
}

static __global__ __launch_bounds__(32, 1) void ec_wt_fp_kernel6(
    const int32_T i1, const int32_T i2, const int32_T nx, int32_T sizeX[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    sizeX[nx] = div_s32_device(i2, i1) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel7(
    const emxArray_real32_T yc, const int32_T lshift_dim0, const int32_T i1,
    const int32_T b, const int32_T c, emxArray_real32_T b_yc, int32_T yc_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    b_i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_yc.data[k + yc_dim0 * b_i1] = yc.data[i1 * k + lshift_dim0 * b_i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel8(
    const emxArray_real32_T yc, const int32_T lshift, emxArray_real32_T b_yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(lshift);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_yc.data[i1] = yc.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel9(
    const int32_T ch, const emxArray_real32_T x, const int32_T b_x,
    emxArray_real32_T c_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_x.data[i1] = x.data[i1 + x_dim0 * ch];
  }
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23])
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23])
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
  nvtxRangePop();
}

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyCpuToGpu_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
  nvtxRangePop();
}

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyGpuToCpu_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  nvtxRangePop();
}

//
// function y = ec_wt_fp(x,fs,lims,voices,ds,doReal,doPwr)
void ec_wt_fp(const emxArray_real32_T *cpu_x, real_T fs, const real_T lims[2],
              real_T voices, real_T ds, boolean_T doReal, boolean_T doPwr,
              emxArray_cell_wrap_0 *y)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  dim3 block;
  dim3 grid;
  emxArray_creal32_T c_gpu_x;
  emxArray_creal32_T c_gpu_yc;
  emxArray_creal32_T d_gpu_yc;
  emxArray_creal32_T gpu_r;
  emxArray_creal32_T gpu_r1;
  emxArray_creal32_T *b_cpu_x;
  emxArray_creal32_T *b_cpu_yc;
  emxArray_creal32_T *c_cpu_yc;
  emxArray_creal32_T *cpu_r;
  emxArray_creal32_T *cpu_r1;
  emxArray_real32_T b_gpu_x;
  emxArray_real32_T b_gpu_yc;
  emxArray_real32_T d_gpu_x;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_y;
  emxArray_real32_T gpu_yc;
  emxArray_real32_T *c_cpu_x;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *cpu_yc;
  emxArray_real32_T *d_cpu_x;
  emxArray_real32_T *d_cpu_yc;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  int32_T(*gpu_lshift)[2];
  int32_T(*gpu_sizeX)[2];
  int32_T b_i;
  int32_T i;
  char_T unusedExpr[23];
  boolean_T b_x_outdatedOnGpu;
  boolean_T r1_outdatedOnCpu;
  boolean_T r1_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnCpu;
  boolean_T x_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_wt_fp#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_lshift, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_sizeX, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_yc);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&d_gpu_yc);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_r1);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&c_gpu_yc);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_yc);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_x);
  r1_outdatedOnCpu = false;
  r1_outdatedOnGpu = false;
  x_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  [electroCUDA] CWT - CUDA mex source
  //    Called by function 'ec_wt' (see for details)
  //    Intended to be compiled into a CUDA mex binary
  //    Kevin Tan, 2024 (github.com/kevmtan/electroCUDA)
  // 'ec_wt_fp:7' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_wt_fp:8' fs (1,1) double
  //  Sampling rate
  // 'ec_wt_fp:9' lims (1,2) double
  //  Frequency limits
  // 'ec_wt_fp:10' voices (1,1) double = 10
  //  Voices per octave
  // 'ec_wt_fp:11' ds (1,1) double = 0
  //  Downsampling factor
  // 'ec_wt_fp:12' doReal (1,1) logical = true
  //  Real output? (complex otherwise)
  // 'ec_wt_fp:13' doPwr (1,1) logical = false
  //  Power output? (magnitude otherwise)
  // 'ec_wt_fp:15' if ds<=1
  if (ds <= 1.0) {
    // 'ec_wt_fp:15' ;
    // 'ec_wt_fp:15' ds=0;
    ds = 0.0;
  }
  //  Prep
  // 'ec_wt_fp:18' coder.gpu.kernelfun;
  //  Trigger CUDA kernel creation
  //  Sizes
  // 'ec_wt_fp:21' nChs = width(x);
  //  Preallocate output
  // 'ec_wt_fp:24' y = coder.nullcopy(cell(1,nChs));
  i = y->size[0] * y->size[1];
  y->size[0] = 1;
  y->size[1] = cpu_x->size[1];
  nvtxMarkA("#emxEnsureCapacity_cell_wrap_0#" MW_AT_LINE);
  emxEnsureCapacity_cell_wrap_0(y, i, &c_emlrtRTEI);
  //  Processing loop across channels
  // 'ec_wt_fp:28' for ch = 1:nChs
  b_i = cpu_x->size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_yc, 2, &p_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&b_cpu_yc, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&b_cpu_x, 2, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_y, 2, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_r, 2, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_r1, 2, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_cpu_x, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_x, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&c_cpu_yc, 2, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_cpu_yc, 2, &k_emlrtRTEI, true);
  profileLoopStart("ec_wt_fp_loop_0", __LINE__, (b_i - 1) + 1, "");
  for (int32_T ch{0}; ch < b_i; ch++) {
    // 'ec_wt_fp:29' if doReal
    if (doReal) {
      int32_T i1;
      int32_T nx;
      uint32_T dv[2];
      boolean_T validLaunchParams;
      // 'ec_wt_fp:30' y{ch} = cwtR_lfn(x(:,ch),fs,lims,voices,ds,doPwr);
      //  Run CWT (magnitude/power)
      //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%% CWT - magnitude out
      // 'ec_wt_fp:55' yc =
      // abs(cwt(xc,"morse",fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices)');
      i = d_cpu_x->size[0];
      d_cpu_x->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(d_cpu_x, i, &e_emlrtRTEI);
      i = cpu_x->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (x_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
      }
      x_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(d_cpu_x, &b_gpu_x, true);
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
      }
      x_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel1#" MW_AT_LINE);
        ec_wt_fp_kernel1<<<grid, block>>>(ch, gpu_x, i, b_gpu_x,
                                          cpu_x->size[0U]);
      }
      b_x_outdatedOnGpu = false;
      nvtxMarkA("#cwt#" MW_AT_LINE);
      coder::cwt(d_cpu_x, &b_gpu_x, &b_x_outdatedOnGpu, fs, lims, voices,
                 cpu_r1, &r1_outdatedOnCpu, &gpu_r1, &r1_outdatedOnGpu);
      i = b_cpu_x->size[0] * b_cpu_x->size[1];
      b_cpu_x->size[0] = cpu_r1->size[1];
      b_cpu_x->size[1] = cpu_r1->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(b_cpu_x, i, &g_emlrtRTEI);
      i = cpu_r1->size[0] - 1;
      nx = cpu_r1->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i, nx), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_r1, &gpu_r1, !r1_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(b_cpu_x, &c_gpu_x, true);
      if (r1_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_r1, cpu_r1);
      }
      r1_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel2#" MW_AT_LINE);
        ec_wt_fp_kernel2<<<grid, block>>>(gpu_r1, nx, i, c_gpu_x,
                                          b_cpu_x->size[0U], cpu_r1->size[0U]);
      }
      nx = b_cpu_x->size[0] * b_cpu_x->size[1];
      profileLoopStart("ec_wt_fp_loop_1", __LINE__, 1 + 1, "");
      for (i1 = 0; i1 < 2; i1++) {
        dv[i1] = static_cast<uint32_T>(b_cpu_x->size[i1]);
      }
      profileLoopEnd();
      i = cpu_yc->size[0] * cpu_yc->size[1];
      cpu_yc->size[0] = static_cast<int32_T>(dv[0]);
      cpu_yc->size[1] = static_cast<int32_T>(dv[1]);
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_yc, i, &i_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_yc, &gpu_yc, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel3#" MW_AT_LINE);
        ec_wt_fp_kernel3<<<grid, block>>>(c_gpu_x, nx - 1, gpu_yc);
      }
      r1_outdatedOnCpu = true;
      //  Convert to power
      // 'ec_wt_fp:58' if doPwr
      if (doPwr) {
        uint32_T yc[2];
        // 'ec_wt_fp:59' yc = yc.^2;
        yc[0] = static_cast<uint32_T>(cpu_yc->size[0]);
        yc[1] = static_cast<uint32_T>(cpu_yc->size[1]);
        profileLoopStart("ec_wt_fp_loop_4", __LINE__, 1 + 1, "");
        for (i1 = 0; i1 < 2; i1++) {
          dv[i1] = yc[i1];
        }
        profileLoopEnd();
        i = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = static_cast<int32_T>(dv[0]);
        cpu_y->size[1] = static_cast<int32_T>(dv[1]);
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_y, i, &l_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(
                static_cast<int32_T>(dv[0]) * static_cast<int32_T>(dv[1]) - 1),
            &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_y, &gpu_y, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel4#" MW_AT_LINE);
          ec_wt_fp_kernel4<<<grid, block>>>(
              gpu_yc,
              static_cast<int32_T>(dv[0]) * static_cast<int32_T>(dv[1]) - 1,
              gpu_y);
        }
        i = cpu_yc->size[0] * cpu_yc->size[1];
        cpu_yc->size[0] = cpu_y->size[0];
        cpu_yc->size[1] = cpu_y->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_yc, i, &m_emlrtRTEI);
        i = cpu_y->size[0] * cpu_y->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_yc, &gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel5#" MW_AT_LINE);
          ec_wt_fp_kernel5<<<grid, block>>>(gpu_y, i, gpu_yc);
        }
      }
      //  Downsample
      // 'ec_wt_fp:63' if ds
      if (ds != 0.0) {
        int32_T cpu_lshift[2];
        int32_T cpu_sizeX[2];
        // 'ec_wt_fp:64' yc = downsample(yc,ds);
        if (!(ds - 1.0 >= 0.0)) {
          d_y = nullptr;
          m2 = emlrtCreateCharArray(2, &iv1[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
          emlrtAssign(&d_y, m2);
          e_y = nullptr;
          m3 = emlrtCreateDoubleScalar(ds - 1.0);
          emlrtAssign(&e_y, m3);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(d_y, e_y, &c_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        nx = 1;
        if (cpu_yc->size[0] != 1) {
          nx = 0;
        }
        r1_outdatedOnGpu = false;
        b_x_outdatedOnGpu = false;
        x_outdatedOnCpu = false;
        profileLoopStart("ec_wt_fp_loop_6", __LINE__, 1 + 1, "");
        for (i1 = 0; i1 < 2; i1++) {
          cpu_sizeX[i1] = cpu_yc->size[i1];
          r1_outdatedOnGpu = true;
          cpu_lshift[i1] = cpu_yc->size[i1];
          x_outdatedOnCpu = true;
        }
        profileLoopEnd();
        nvtxMarkA("#circshift#" MW_AT_LINE);
        coder::circshift(cpu_lshift, &b_x_outdatedOnGpu, *gpu_lshift,
                         &x_outdatedOnCpu, -nx);
        if (b_x_outdatedOnGpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if ((ds > 0.0) && (cpu_lshift[0] < 1)) {
          i1 = 1;
          i = -1;
        } else {
          i1 = static_cast<int32_T>(ds);
          i = cpu_lshift[0] - 1;
        }
        if (r1_outdatedOnGpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(*gpu_sizeX, cpu_sizeX, 8UL, hipMemcpyHostToDevice),
              __FILE__, __LINE__);
        }
        nvtxMarkA("#ec_wt_fp_kernel6#" MW_AT_LINE);
        ec_wt_fp_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(i1, i, nx,
                                                                  *gpu_sizeX);
        nx = d_cpu_yc->size[0] * d_cpu_yc->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        d_cpu_yc->size[0] = div_s32(i, i1) + 1;
        d_cpu_yc->size[1] = cpu_lshift[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(d_cpu_yc, nx, &k_emlrtRTEI);
        i /= i1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, i), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(d_cpu_yc, &b_gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel7#" MW_AT_LINE);
          ec_wt_fp_kernel7<<<grid, block>>>(gpu_yc, cpu_lshift[0], i1, i,
                                            cpu_lshift[1] - 1, b_gpu_yc,
                                            d_cpu_yc->size[0U]);
        }
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(
            hipMemcpy(cpu_sizeX, *gpu_sizeX, 8UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
        i = cpu_yc->size[0] * cpu_yc->size[1];
        cpu_yc->size[0] = cpu_sizeX[0];
        cpu_yc->size[1] = cpu_sizeX[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_yc, i, &o_emlrtRTEI);
        i = cpu_sizeX[0] * cpu_sizeX[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_yc, &gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel8#" MW_AT_LINE);
          ec_wt_fp_kernel8<<<grid, block>>>(b_gpu_yc, i, gpu_yc);
        }
      }
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = cpu_yc->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(y->data[ch].f1, i, &j_emlrtRTEI);
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = cpu_yc->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(y->data[ch].f1, i, &j_emlrtRTEI);
      profileLoopStart("ec_wt_fp_loop_5", __LINE__,
                       (cpu_yc->size[0] * cpu_yc->size[1] - 1) + 1, "");
      for (i = 0; i < cpu_yc->size[0] * cpu_yc->size[1]; i++) {
        if (r1_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_yc, &gpu_yc);
        }
        r1_outdatedOnCpu = false;
        y->data[ch].f1->data[i].re = cpu_yc->data[i];
        y->data[ch].f1->data[i].im = 0.0F;
      }
      profileLoopEnd();
    } else {
      int32_T nx;
      boolean_T validLaunchParams;
      // 'ec_wt_fp:31' else
      // 'ec_wt_fp:32' y{ch} = cwt_lfn(x(:,ch),fs,lims,voices,ds);
      //  Run CWT (complex)
      //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%% CWT
      // 'ec_wt_fp:43' yc =
      // cwt(xc,"morse",fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices)';
      i = c_cpu_x->size[0];
      c_cpu_x->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(c_cpu_x, i, &d_emlrtRTEI);
      i = cpu_x->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      if (x_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
      }
      x_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(c_cpu_x, &d_gpu_x, true);
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
      }
      x_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel9#" MW_AT_LINE);
        ec_wt_fp_kernel9<<<grid, block>>>(ch, gpu_x, i, d_gpu_x,
                                          cpu_x->size[0U]);
      }
      b_x_outdatedOnGpu = false;
      nvtxMarkA("#cwt#" MW_AT_LINE);
      coder::cwt(c_cpu_x, &d_gpu_x, &b_x_outdatedOnGpu, fs, lims, voices, cpu_r,
                 &r1_outdatedOnCpu, &gpu_r, &r1_outdatedOnGpu);
      i = b_cpu_yc->size[0] * b_cpu_yc->size[1];
      b_cpu_yc->size[0] = cpu_r->size[1];
      b_cpu_yc->size[1] = cpu_r->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(b_cpu_yc, i, &f_emlrtRTEI);
      i = cpu_r->size[0] - 1;
      nx = cpu_r->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i, nx), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(cpu_r, &gpu_r, !r1_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal32_T(b_cpu_yc, &c_gpu_yc, true);
      if (r1_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_r, cpu_r);
      }
      r1_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel10#" MW_AT_LINE);
        ec_wt_fp_kernel10<<<grid, block>>>(gpu_r, nx, i, c_gpu_yc,
                                           b_cpu_yc->size[0U], cpu_r->size[0U]);
      }
      r1_outdatedOnCpu = true;
      //  Downsample
      // 'ec_wt_fp:46' if ds
      if (ds != 0.0) {
        int32_T cpu_lshift[2];
        int32_T cpu_sizeX[2];
        int32_T i1;
        // 'ec_wt_fp:47' yc = downsample(yc,ds);
        if (!(ds - 1.0 >= 0.0)) {
          b_y = nullptr;
          m = emlrtCreateCharArray(2, &iv[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
          emlrtAssign(&b_y, m);
          c_y = nullptr;
          m1 = emlrtCreateDoubleScalar(ds - 1.0);
          emlrtAssign(&c_y, m1);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(b_y, c_y, &c_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        nx = 1;
        if (b_cpu_yc->size[0] != 1) {
          nx = 0;
        }
        r1_outdatedOnGpu = false;
        b_x_outdatedOnGpu = false;
        x_outdatedOnCpu = false;
        profileLoopStart("ec_wt_fp_loop_3", __LINE__, 1 + 1, "");
        for (i1 = 0; i1 < 2; i1++) {
          cpu_sizeX[i1] = b_cpu_yc->size[i1];
          r1_outdatedOnGpu = true;
          cpu_lshift[i1] = b_cpu_yc->size[i1];
          x_outdatedOnCpu = true;
        }
        profileLoopEnd();
        nvtxMarkA("#circshift#" MW_AT_LINE);
        coder::circshift(cpu_lshift, &b_x_outdatedOnGpu, *gpu_lshift,
                         &x_outdatedOnCpu, -nx);
        if (b_x_outdatedOnGpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if ((ds > 0.0) && (cpu_lshift[0] < 1)) {
          i1 = 1;
          i = -1;
        } else {
          i1 = static_cast<int32_T>(ds);
          i = cpu_lshift[0] - 1;
        }
        if (r1_outdatedOnGpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(*gpu_sizeX, cpu_sizeX, 8UL, hipMemcpyHostToDevice),
              __FILE__, __LINE__);
        }
        nvtxMarkA("#ec_wt_fp_kernel11#" MW_AT_LINE);
        ec_wt_fp_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(i1, i, nx,
                                                                   *gpu_sizeX);
        nx = c_cpu_yc->size[0] * c_cpu_yc->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        c_cpu_yc->size[0] = div_s32(i, i1) + 1;
        c_cpu_yc->size[1] = cpu_lshift[1];
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(c_cpu_yc, nx, &k_emlrtRTEI);
        i /= i1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, i), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(c_cpu_yc, &d_gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel12#" MW_AT_LINE);
          ec_wt_fp_kernel12<<<grid, block>>>(c_gpu_yc, cpu_lshift[0], i1, i,
                                             cpu_lshift[1] - 1, d_gpu_yc,
                                             c_cpu_yc->size[0U]);
        }
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(
            hipMemcpy(cpu_sizeX, *gpu_sizeX, 8UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
        i = b_cpu_yc->size[0] * b_cpu_yc->size[1];
        b_cpu_yc->size[0] = cpu_sizeX[0];
        b_cpu_yc->size[1] = cpu_sizeX[1];
        nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
        emxEnsureCapacity_creal32_T(b_cpu_yc, i, &n_emlrtRTEI);
        i = cpu_sizeX[0] * cpu_sizeX[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal32_T(b_cpu_yc, &c_gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel13#" MW_AT_LINE);
          ec_wt_fp_kernel13<<<grid, block>>>(d_gpu_yc, i, c_gpu_yc);
        }
      }
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = b_cpu_yc->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(y->data[ch].f1, i, &h_emlrtRTEI);
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = b_cpu_yc->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
      emxEnsureCapacity_creal32_T(y->data[ch].f1, i, &h_emlrtRTEI);
      profileLoopStart("ec_wt_fp_loop_2", __LINE__,
                       (b_cpu_yc->size[0] * b_cpu_yc->size[1] - 1) + 1, "");
      for (i = 0; i < b_cpu_yc->size[0] * b_cpu_yc->size[1]; i++) {
        if (r1_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal32_T(b_cpu_yc, &c_gpu_yc);
        }
        r1_outdatedOnCpu = false;
        y->data[ch].f1->data[i] = b_cpu_yc->data[i];
      }
      profileLoopEnd();
    }
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_yc);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&c_cpu_yc);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_cpu_x);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_cpu_x);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_r1);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_r);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_y);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&b_cpu_x);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&b_cpu_yc);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_yc);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_yc);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&c_gpu_yc);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_r1);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&d_gpu_yc);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_yc);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_sizeX), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_lshift), __FILE__, __LINE__);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_wt_fp.cu)

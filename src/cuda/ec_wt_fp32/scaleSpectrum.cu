#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// scaleSpectrum.cu
//
// Code generation for function 'scaleSpectrum'
//

// Include files
#include "scaleSpectrum.h"
#include "div.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "numCpsi.h"
#include "rt_nonfinite.h"
#include "vvarstd.h"
#include "wt.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo ed_emlrtRTEI{
    12,                                                               // lineNo
    5,                                                                // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    211,                                                              // lineNo
    24,                                                               // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo gd_emlrtRTEI{
    96,                                                               // lineNo
    5,                                                                // colNo
    "trapz",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/trapz.m" // pName
};

static emlrtRTEInfo id_emlrtRTEI{
    298,             // lineNo
    18,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo jd_emlrtRTEI{
    297,             // lineNo
    5,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo kd_emlrtRTEI{
    12,            // lineNo
    12,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo ld_emlrtRTEI{
    22,            // lineNo
    32,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo md_emlrtRTEI{
    12,            // lineNo
    1,             // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo nd_emlrtRTEI{
    26,            // lineNo
    43,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo od_emlrtRTEI{
    26,            // lineNo
    37,            // colNo
    "scNormalize", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "scNormalize.m" // pName
};

static emlrtRTEInfo pd_emlrtRTEI{
    293,             // lineNo
    14,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

// Function Declarations
static __global__ void c_cwtfilterbank_scaleSpectrum_k(const real32_T c1,
                                                       emxArray_real32_T c);

static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line);

static __global__ void d_cwtfilterbank_scaleSpectrum_k(
    const real32_T c1, const uint32_T defaultSL_idx_1, emxArray_real32_T c);

static __global__ void e_cwtfilterbank_scaleSpectrum_k(const real32_T c1,
                                                       emxArray_real32_T c);

static __global__ void f_cwtfilterbank_scaleSpectrum_k(const real32_T c1,
                                                       emxArray_real32_T c,
                                                       int32_T z_dim0);

static real32_T rt_hypotf_snf(real32_T u0, real32_T u1);

// Function Definitions
static __global__ __launch_bounds__(32, 1) void c_cwtfilterbank_scaleSpectrum_k(
    const real32_T c1, emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = c1;
  }
}

static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line)
{
  const char *errName;
  const char *errString;
  nvtxRangePushA("#fcn#cublasCheck#" MW_AT_LOCATION);
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    nvtxMarkA("#raiseCudaError#" MW_AT_LINE);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
  nvtxRangePop();
}

static __global__ __launch_bounds__(32, 1) void d_cwtfilterbank_scaleSpectrum_k(
    const real32_T c1, const uint32_T defaultSL_idx_1, emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[static_cast<int32_T>(defaultSL_idx_1) - 1] = c1;
  }
}

static __global__ __launch_bounds__(32, 1) void e_cwtfilterbank_scaleSpectrum_k(
    const real32_T c1, emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = c1;
  }
}

static __global__ __launch_bounds__(32, 1) void f_cwtfilterbank_scaleSpectrum_k(
    const real32_T c1, emxArray_real32_T c, int32_T z_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[z_dim0 - 1] = c1;
  }
}

static real32_T rt_hypotf_snf(real32_T u0, real32_T u1)
{
  real32_T a;
  real32_T b;
  real32_T y;
  nvtxRangePushA("#fcn#rt_hypotf_snf#" MW_AT_LOCATION);
  a = std::abs(u0);
  b = std::abs(u1);
  if (a < b) {
    a /= b;
    y = b * std::sqrt(a * a + 1.0F);
  } else if (a > b) {
    b /= a;
    y = a * std::sqrt(b * b + 1.0F);
  } else if (std::isnan(b)) {
    y = rtNaNF;
  } else {
    y = a * 1.41421354F;
  }
  nvtxRangePop();
  return y;
}

//
//
namespace coder {
void cwtfilterbank_scaleSpectrum(cwtfilterbank *self,
                                 const emxArray_real32_T *x,
                                 emxArray_real32_T *cpu_savgp,
                                 boolean_T *savgp_outdatedOnCpu,
                                 emxArray_real32_T *gpu_savgp,
                                 boolean_T *savgp_outdatedOnGpu)
{
  emxArray_creal32_T *cfs;
  emxArray_real32_T b_gpu_c;
  emxArray_real32_T gpu_c;
  emxArray_real32_T gpu_y;
  emxArray_real32_T gpu_z;
  emxArray_real32_T *Scales;
  emxArray_real32_T *a;
  emxArray_real32_T *abscfssq;
  emxArray_real32_T *abswt2S;
  emxArray_real32_T *b_cpu_c;
  emxArray_real32_T *c_y;
  emxArray_real32_T *cpu_c;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *cpu_z;
  emxArray_real32_T *d_y;
  emxArray_real32_T *y;
  emxArray_real_T *c;
  emxArray_uint32_T *b_y;
  int32_T nx;
  int32_T u1;
  int32_T vstride;
  real32_T alpha1;
  real32_T b_c1;
  real32_T cpu_b;
  real32_T normfac;
  real32_T *gpu_b;
  uint32_T defaultSL_idx_1;
  boolean_T b_outdatedOnCpu;
  boolean_T c_needsGpuEnsureCapacity;
  boolean_T c_outdatedOnGpu;
  boolean_T z_needsGpuEnsureCapacity;
  boolean_T z_outdatedOnGpu;
  nvtxRangePushA("#fcn#cwtfilterbank_scaleSpectrum#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_c);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_c);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_b, 4UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_z);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  defaultSL_idx_1 = static_cast<uint32_T>(self->Scales->size[1]);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&Scales, 2, &e_emlrtRTEI, true);
  vstride = Scales->size[0] * Scales->size[1];
  Scales->size[0] = 1;
  Scales->size[1] = self->Scales->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(Scales, vstride, &e_emlrtRTEI);
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_0", __LINE__,
                   (self->Scales->size[1] - 1) + 1, "");
  for (vstride = 0; vstride < self->Scales->size[1]; vstride++) {
    Scales->data[vstride] = static_cast<real32_T>(self->Scales->data[vstride]);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cfs, 2, &ab_emlrtRTEI, true);
  nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
  cwtfilterbank_wt(self, x, cfs);
  nvtxMarkA("#vvarstd#" MW_AT_LINE);
  normfac = vvarstd(x, x->size[0]);
  nx = cfs->size[0] * cfs->size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&y, 2, &kd_emlrtRTEI, true);
  vstride = y->size[0] * y->size[1];
  y->size[0] = cfs->size[0];
  y->size[1] = cfs->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(y, vstride, &j_emlrtRTEI);
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_1", __LINE__, (nx - 1) + 1,
                   "");
  for (int32_T k{0}; k < nx; k++) {
    nvtxMarkA("#rt_hypotf_snf#" MW_AT_LINE);
    y->data[k] = rt_hypotf_snf(cfs->data[k].re, cfs->data[k].im);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&a, 2, &ld_emlrtRTEI, true);
  vstride = a->size[0] * a->size[1];
  a->size[0] = y->size[0];
  a->size[1] = y->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(a, vstride, &m_emlrtRTEI);
  nx = y->size[0] * y->size[1];
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_2", __LINE__, (nx - 1) + 1,
                   "");
  for (int32_T k{0}; k < nx; k++) {
    a->data[k] = y->data[k] * y->data[k];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&y);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&abscfssq, 2, &md_emlrtRTEI, true);
  vstride = abscfssq->size[0] * abscfssq->size[1];
  nx = Scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (Scales->size[1] == 1) {
    abscfssq->size[0] = a->size[0];
  } else if (a->size[0] == 1) {
    abscfssq->size[0] = Scales->size[1];
  } else if (a->size[0] == Scales->size[1]) {
    abscfssq->size[0] = a->size[0];
  } else {
    abscfssq->size[0] = u1;
  }
  abscfssq->size[1] = a->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(abscfssq, vstride, &p_emlrtRTEI);
  nx = Scales->size[1];
  u1 = a->size[0];
  if (nx <= u1) {
    u1 = nx;
  }
  if (Scales->size[1] == 1) {
    u1 = a->size[0];
  } else if (a->size[0] == 1) {
    u1 = Scales->size[1];
  } else if (a->size[0] == Scales->size[1]) {
    u1 = a->size[0];
  }
  if ((u1 != 0) && (a->size[1] != 0)) {
    int32_T acoef;
    int32_T bcoef;
    nx = (a->size[1] != 1);
    u1 = a->size[1] - 1;
    acoef = (a->size[0] != 1);
    bcoef = (Scales->size[1] != 1);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_3", __LINE__, u1 + 1,
                     "");
    for (int32_T k{0}; k <= u1; k++) {
      int32_T i1;
      vstride = nx * k;
      i1 = abscfssq->size[0] - 1;
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_4", __LINE__, i1 + 1,
                       "");
      for (int32_T b_k{0}; b_k <= i1; b_k++) {
        abscfssq->data[b_k + abscfssq->size[0] * k] =
            a->data[acoef * b_k + a->size[0] * vstride] /
            Scales->data[bcoef * b_k];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&a);
  nvtxMarkA("#emxInit_uint32_T#" MW_AT_LINE);
  emxInit_uint32_T(&b_y, 2, &nd_emlrtRTEI, true);
  if (abscfssq->size[1] < 1) {
    b_y->size[0] = 1;
    b_y->size[1] = 0;
  } else {
    uint32_T u;
    u = static_cast<uint32_T>(abscfssq->size[1]);
    vstride = b_y->size[0] * b_y->size[1];
    b_y->size[0] = 1;
    b_y->size[1] = abscfssq->size[1];
    nvtxMarkA("#emxEnsureCapacity_uint32_T#" MW_AT_LINE);
    emxEnsureCapacity_uint32_T(b_y, vstride, &ib_emlrtRTEI);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_5", __LINE__,
                     (static_cast<int32_T>(u) - 1) + 1, "");
    for (vstride = 0; vstride < static_cast<int32_T>(u); vstride++) {
      b_y->data[vstride] = static_cast<uint32_T>(vstride) + 1U;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_z, 1, &od_emlrtRTEI, true);
  z_outdatedOnGpu = false;
  vstride = cpu_z->size[0];
  cpu_z->size[0] = abscfssq->size[0];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(cpu_z, vstride, &ed_emlrtRTEI);
  z_needsGpuEnsureCapacity = true;
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_6", __LINE__,
                   (abscfssq->size[0] - 1) + 1, "");
  for (vstride = 0; vstride < abscfssq->size[0]; vstride++) {
    cpu_z->data[vstride] = 0.0F;
    z_outdatedOnGpu = true;
  }
  profileLoopEnd();
  if (abscfssq->size[1] > 1) {
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&c, 1, &gd_emlrtRTEI, true);
    if (b_y->size[1] == 1) {
      nx = abscfssq->size[1];
      vstride = c->size[0];
      c->size[0] = abscfssq->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(c, vstride, &gd_emlrtRTEI);
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_7", __LINE__,
                       (nx - 1) + 1, "");
      for (vstride = 0; vstride < nx; vstride++) {
        c->data[vstride] = b_y->data[0];
      }
      real_T c1;
      profileLoopEnd();
      c1 = 0.5 * static_cast<real_T>(b_y->data[0]);
      c->data[0] = c1;
      c->data[abscfssq->size[1] - 1] = c1;
    } else {
      vstride = c->size[0];
      c->size[0] = abscfssq->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(c, vstride, &fd_emlrtRTEI);
      c->data[0] =
          0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[1]) -
                                    static_cast<int32_T>(b_y->data[0]));
      nx = abscfssq->size[1];
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_8", __LINE__,
                       (nx - 3) + 1, "");
      for (int32_T k{0}; k <= nx - 3; k++) {
        c->data[k + 1] =
            0.5 * static_cast<real_T>(static_cast<int32_T>(b_y->data[k + 2]) -
                                      static_cast<int32_T>(b_y->data[k]));
      }
      profileLoopEnd();
      c->data[abscfssq->size[1] - 1] =
          0.5 * static_cast<real_T>(
                    static_cast<int32_T>(b_y->data[abscfssq->size[1] - 1]) -
                    static_cast<int32_T>(b_y->data[abscfssq->size[1] - 2]));
    }
    vstride = abscfssq->size[0];
    if (abscfssq->size[0] != 0) {
      int64_T b;
      nx = abscfssq->size[0];
      std::memset(&cpu_z->data[0], 0,
                  static_cast<uint32_T>(nx) * sizeof(real32_T));
      z_outdatedOnGpu = true;
      nx = 0;
      u1 = abscfssq->size[0] * (abscfssq->size[1] - 1) + 1;
      nvtxMarkA("#computeEndIdx#" MW_AT_LINE);
      b = computeEndIdx(1L, static_cast<int64_T>(u1),
                        static_cast<int64_T>(vstride));
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_13", __LINE__, b + 1L,
                       "");
      for (int64_T iac{0L}; iac <= b; iac++) {
        int64_T d;
        nvtxMarkA("#computeEndIdx#" MW_AT_LINE);
        d = computeEndIdx(
            static_cast<int64_T>(static_cast<int32_T>(iac * vstride + 1L)),
            static_cast<int64_T>(
                (static_cast<int32_T>(iac * vstride + 1L) + vstride) - 1),
            1L);
        profileLoopStart("cwtfilterbank_scaleSpectrum_loop_14", __LINE__,
                         d + 1L, "");
        for (int64_T ia{0L}; ia <= d; ia++) {
          cpu_z->data[static_cast<int32_T>(ia)] +=
              abscfssq
                  ->data[static_cast<int32_T>(
                             static_cast<int32_T>(iac * vstride + 1L) + ia) -
                         1] *
              static_cast<real32_T>(c->data[nx]);
        }
        profileLoopEnd();
        nx++;
      }
      profileLoopEnd();
    }
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&c);
  }
  nvtxMarkA("#emxFree_uint32_T#" MW_AT_LINE);
  emxFree_uint32_T(&b_y);
  cpu_b = 0.0F;
  b_outdatedOnCpu = false;
  if (cpu_z->size[0] <= 1) {
    if ((Scales->size[1] == 1) &&
        (std::isinf(Scales->data[0]) || std::isnan(Scales->data[0]))) {
      cpu_b = rtNaNF;
    }
  } else {
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&cpu_c, 1, &gd_emlrtRTEI, true);
    if (Scales->size[1] == 1) {
      nx = cpu_z->size[0];
      c_outdatedOnGpu = false;
      vstride = cpu_c->size[0];
      cpu_c->size[0] = cpu_z->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_c, vstride, &gd_emlrtRTEI);
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_10", __LINE__,
                       (nx - 1) + 1, "");
      for (vstride = 0; vstride < nx; vstride++) {
        cpu_c->data[vstride] = Scales->data[0];
        c_outdatedOnGpu = true;
      }
      profileLoopEnd();
      b_c1 = 0.5F * Scales->data[0];
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_c, &b_gpu_c, !c_outdatedOnGpu);
      if (c_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&b_gpu_c, cpu_c);
      }
      nvtxMarkA("#e_cwtfilterbank_scaleSpectrum_k#" MW_AT_LINE);
      e_cwtfilterbank_scaleSpectrum_k<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          b_c1, b_gpu_c);
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
      z_needsGpuEnsureCapacity = false;
      c_needsGpuEnsureCapacity = false;
      if (z_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_z, cpu_z);
      }
      z_outdatedOnGpu = false;
      nvtxMarkA("#f_cwtfilterbank_scaleSpectrum_k#" MW_AT_LINE);
      f_cwtfilterbank_scaleSpectrum_k<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          b_c1, b_gpu_c, cpu_z->size[0U]);
      c_outdatedOnGpu = false;
    } else {
      vstride = cpu_c->size[0];
      cpu_c->size[0] = cpu_z->size[0];
      nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
      emxEnsureCapacity_real32_T(cpu_c, vstride, &fd_emlrtRTEI);
      cpu_c->data[0] = 0.5F * (Scales->data[1] - Scales->data[0]);
      nx = cpu_z->size[0];
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_9", __LINE__,
                       (nx - 3) + 1, "");
      for (int32_T k{0}; k <= nx - 3; k++) {
        cpu_c->data[k + 1] = 0.5F * (Scales->data[k + 2] - Scales->data[k]);
      }
      profileLoopEnd();
      cpu_c->data[cpu_z->size[0] - 1] =
          0.5F *
          (Scales->data[cpu_z->size[0] - 1] - Scales->data[cpu_z->size[0] - 2]);
      c_outdatedOnGpu = true;
      c_needsGpuEnsureCapacity = true;
    }
    b_c1 = 0.0F;
    alpha1 = 1.0F;
    if (z_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_z, &gpu_z, !z_outdatedOnGpu);
    }
    if (c_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real32_T(cpu_c, &b_gpu_c, !c_outdatedOnGpu);
    }
    if (z_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_z, cpu_z);
    }
    if (c_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&b_gpu_c, cpu_c);
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(hipMemcpy(gpu_b, &cpu_b, 4UL, hipMemcpyHostToDevice),
                   __FILE__, __LINE__);
    nvtxMarkA("#cublasCheck#" MW_AT_LINE);
    cublasCheck(hipblasSgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                            cpu_z->size[0], 1, (float *)&alpha1,
                            (float *)&gpu_z.data[0], cpu_z->size[0],
                            (float *)&b_gpu_c.data[0], 1, (float *)&b_c1,
                            (float *)gpu_b, 1),
                __FILE__, __LINE__);
    b_outdatedOnCpu = true;
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&cpu_c);
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_z);
  if (!(normfac == 0.0F)) {
    if (b_outdatedOnCpu) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(hipMemcpy(&cpu_b, gpu_b, 4UL, hipMemcpyDeviceToHost),
                     __FILE__, __LINE__);
    }
    nvtxMarkA("#numCpsi#" MW_AT_LINE);
    b_c1 = std::sqrt(
        normfac /
        (static_cast<real32_T>(2.0 /
                               wavelet::internal::cwt::numCpsi(
                                   self->Wavelet, self->Gamma, self->Beta) /
                               static_cast<real_T>(abscfssq->size[1])) *
         cpu_b));
    vstride = cfs->size[0] * cfs->size[1];
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_11", __LINE__,
                     (vstride - 1) + 1, "");
    for (u1 = 0; u1 < vstride; u1++) {
      cfs->data[u1].re *= b_c1;
      cfs->data[u1].im *= b_c1;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&abscfssq);
  nx = cfs->size[0] * cfs->size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&c_y, 2, &pd_emlrtRTEI, true);
  vstride = c_y->size[0] * c_y->size[1];
  c_y->size[0] = cfs->size[0];
  c_y->size[1] = cfs->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(c_y, vstride, &j_emlrtRTEI);
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_12", __LINE__,
                   (nx - 1) + 1, "");
  for (int32_T k{0}; k < nx; k++) {
    nvtxMarkA("#rt_hypotf_snf#" MW_AT_LINE);
    c_y->data[k] = rt_hypotf_snf(cfs->data[k].re, cfs->data[k].im);
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cfs);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&d_y, 2, &pd_emlrtRTEI, true);
  vstride = d_y->size[0] * d_y->size[1];
  d_y->size[0] = c_y->size[0];
  d_y->size[1] = c_y->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(d_y, vstride, &m_emlrtRTEI);
  nx = c_y->size[0] * c_y->size[1];
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_15", __LINE__,
                   (nx - 1) + 1, "");
  for (int32_T k{0}; k < nx; k++) {
    d_y->data[k] = c_y->data[k] * c_y->data[k];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&c_y);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&abswt2S, 2, &hd_emlrtRTEI, true);
  if (d_y->size[0] == Scales->size[1]) {
    vstride = abswt2S->size[0] * abswt2S->size[1];
    abswt2S->size[0] = d_y->size[0];
    abswt2S->size[1] = d_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(abswt2S, vstride, &hd_emlrtRTEI);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_16", __LINE__,
                     (d_y->size[1] - 1) + 1, "");
    for (vstride = 0; vstride < d_y->size[1]; vstride++) {
      profileLoopStart("cwtfilterbank_scaleSpectrum_loop_17", __LINE__,
                       (d_y->size[0] - 1) + 1, "");
      for (u1 = 0; u1 < d_y->size[0]; u1++) {
        abswt2S->data[u1 + abswt2S->size[0] * vstride] =
            d_y->data[u1 + d_y->size[0] * vstride] / Scales->data[u1];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  } else {
    nvtxMarkA("#binary_expand_op_6#" MW_AT_LINE);
    binary_expand_op_6(abswt2S, d_y, Scales);
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&d_y);
  nx = static_cast<int32_T>(defaultSL_idx_1);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_y, 2, &id_emlrtRTEI, true);
  b_outdatedOnCpu = false;
  vstride = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = static_cast<int32_T>(defaultSL_idx_1);
  cpu_y->size[1] = abswt2S->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(cpu_y, vstride, &id_emlrtRTEI);
  profileLoopStart("cwtfilterbank_scaleSpectrum_loop_18", __LINE__,
                   (abswt2S->size[1] - 1) + 1, "");
  for (vstride = 0; vstride < abswt2S->size[1]; vstride++) {
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_19", __LINE__,
                     (nx - 1) + 1, "");
    for (u1 = 0; u1 < nx; u1++) {
      cpu_y->data[u1 + cpu_y->size[0] * vstride] =
          abswt2S->data[u1 + abswt2S->size[0] * vstride];
      b_outdatedOnCpu = true;
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_c, 1, &gd_emlrtRTEI, true);
  if ((static_cast<int32_T>(defaultSL_idx_1) == 0) && (abswt2S->size[1] == 0)) {
    if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
      b_c1 = Scales->data[0] * 0.0F;
    } else {
      b_c1 = 0.0F;
    }
    vstride = cpu_savgp->size[0] * cpu_savgp->size[1];
    cpu_savgp->size[0] = 1;
    cpu_savgp->size[1] = 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_savgp, vstride, &jd_emlrtRTEI);
    cpu_savgp->data[0] = b_c1;
    *savgp_outdatedOnCpu = false;
    *savgp_outdatedOnGpu = true;
  } else {
    *savgp_outdatedOnCpu = false;
    *savgp_outdatedOnGpu = false;
    vstride = cpu_savgp->size[0] * cpu_savgp->size[1];
    cpu_savgp->size[0] = 1;
    cpu_savgp->size[1] = abswt2S->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_savgp, vstride, &jd_emlrtRTEI);
    profileLoopStart("cwtfilterbank_scaleSpectrum_loop_20", __LINE__,
                     (abswt2S->size[1] - 1) + 1, "");
    for (vstride = 0; vstride < abswt2S->size[1]; vstride++) {
      cpu_savgp->data[vstride] = 0.0F;
      *savgp_outdatedOnGpu = true;
    }
    profileLoopEnd();
    if (static_cast<int32_T>(defaultSL_idx_1) <= 1) {
      if ((static_cast<int32_T>(defaultSL_idx_1) == 1) &&
          (std::isinf(Scales->data[0]) || std::isnan(Scales->data[0]))) {
        *savgp_outdatedOnGpu = false;
        vstride = cpu_savgp->size[0] * cpu_savgp->size[1];
        cpu_savgp->size[0] = 1;
        cpu_savgp->size[1] = abswt2S->size[1];
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(cpu_savgp, vstride, &jd_emlrtRTEI);
        profileLoopStart("cwtfilterbank_scaleSpectrum_loop_23", __LINE__,
                         (abswt2S->size[1] - 1) + 1, "");
        for (vstride = 0; vstride < abswt2S->size[1]; vstride++) {
          cpu_savgp->data[vstride] = rtNaNF;
          *savgp_outdatedOnGpu = true;
        }
        profileLoopEnd();
      }
    } else {
      if (static_cast<int32_T>(defaultSL_idx_1) == 1) {
        c_outdatedOnGpu = false;
        vstride = b_cpu_c->size[0];
        b_cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_cpu_c, vstride, &gd_emlrtRTEI);
        profileLoopStart("cwtfilterbank_scaleSpectrum_loop_22", __LINE__,
                         (nx - 1) + 1, "");
        for (vstride = 0; vstride < nx; vstride++) {
          b_cpu_c->data[vstride] = Scales->data[0];
          c_outdatedOnGpu = true;
        }
        profileLoopEnd();
        b_c1 = 0.5F * Scales->data[0];
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(b_cpu_c, &gpu_c, !c_outdatedOnGpu);
        if (c_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_c, b_cpu_c);
        }
        nvtxMarkA("#c_cwtfilterbank_scaleSpectrum_k#" MW_AT_LINE);
        c_cwtfilterbank_scaleSpectrum_k<<<dim3(1U, 1U, 1U),
                                          dim3(32U, 1U, 1U)>>>(b_c1, gpu_c);
        c_needsGpuEnsureCapacity = false;
        nvtxMarkA("#d_cwtfilterbank_scaleSpectrum_k#" MW_AT_LINE);
        d_cwtfilterbank_scaleSpectrum_k<<<dim3(1U, 1U, 1U),
                                          dim3(32U, 1U, 1U)>>>(
            b_c1, defaultSL_idx_1, gpu_c);
        c_outdatedOnGpu = false;
      } else {
        vstride = b_cpu_c->size[0];
        b_cpu_c->size[0] = static_cast<int32_T>(defaultSL_idx_1);
        nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
        emxEnsureCapacity_real32_T(b_cpu_c, vstride, &fd_emlrtRTEI);
        b_cpu_c->data[0] = 0.5F * (Scales->data[1] - Scales->data[0]);
        profileLoopStart("cwtfilterbank_scaleSpectrum_loop_21", __LINE__,
                         (nx - 3) + 1, "");
        for (int32_T k{0}; k <= nx - 3; k++) {
          b_cpu_c->data[k + 1] = 0.5F * (Scales->data[k + 2] - Scales->data[k]);
        }
        profileLoopEnd();
        b_cpu_c->data[static_cast<int32_T>(defaultSL_idx_1) - 1] =
            0.5F * (Scales->data[static_cast<int32_T>(defaultSL_idx_1) - 1] -
                    Scales->data[static_cast<int32_T>(defaultSL_idx_1) - 2]);
        c_outdatedOnGpu = true;
        c_needsGpuEnsureCapacity = true;
      }
      if (abswt2S->size[1] >= 1) {
        b_c1 = 0.0F;
        alpha1 = 1.0F;
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_y, &gpu_y, !b_outdatedOnCpu);
        if (c_needsGpuEnsureCapacity) {
          nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real32_T(b_cpu_c, &gpu_c, !c_outdatedOnGpu);
        }
        nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real32_T(cpu_savgp, gpu_savgp,
                                      !*savgp_outdatedOnGpu);
        if (b_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_y, cpu_y);
        }
        if (c_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_c, b_cpu_c);
        }
        if (*savgp_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_savgp, cpu_savgp);
        }
        nvtxMarkA("#cublasCheck#" MW_AT_LINE);
        cublasCheck(hipblasSgemv(getCublasGlobalHandle(), HIPBLAS_OP_T,
                                static_cast<int32_T>(defaultSL_idx_1),
                                abswt2S->size[1], (float *)&alpha1,
                                (float *)&gpu_y.data[0],
                                static_cast<int32_T>(defaultSL_idx_1),
                                (float *)&gpu_c.data[0], 1, (float *)&b_c1,
                                (float *)&gpu_savgp->data[0], 1),
                    __FILE__, __LINE__);
        *savgp_outdatedOnGpu = false;
        *savgp_outdatedOnCpu = true;
      }
    }
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_c);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&abswt2S);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&Scales);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_z);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(gpu_b), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_c);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_c);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (scaleSpectrum.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp_mexutil.cu
//
// Code generation for function 'ec_wt_fp_mexutil'
//

// Include files
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cmath>
#include <cstdlib>
#include <cstring>

// Function Definitions
void b_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *msgId,
                        char_T ret[23])
{
  static const int32_T dims[2]{1, 23};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 23);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

void b_raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                      const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  nvtxRangePushA("#fcn#b_raiseCudaError#" MW_AT_LOCATION);
  len = strlen(file);
  pn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  fn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                         emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  const mxArray *m3;
  nvtxRangePushA("#fcn#b_sprintf#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  m3 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                             "sprintf", true, location);
  nvtxRangePop();
  return m3;
}

void checkCudaError(hipError_t errCode, const char_T *file, uint32_T b_line)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errCode != hipSuccess) {
    nvtxMarkA("#b_raiseCudaError#" MW_AT_LINE);
    b_raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                     hipGetErrorString(errCode));
  }
  nvtxRangePop();
}

int64_T computeEndIdx(int64_T start, int64_T end, int64_T stride)
{
  int64_T newEnd;
  nvtxRangePushA("#fcn#computeEndIdx#" MW_AT_LOCATION);
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  nvtxRangePop();
  return newEnd;
}

uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  nvtxRangePop();
  return numIters;
}

uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
  }
  numIters *= n;
  nvtxRangePop();
  return numIters;
}

real_T d_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  nvtxRangePushA("#fcn#d_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 0U,
                          (const void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  nvtxRangePop();
  return ret;
}

int32_T div_s32(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  nvtxRangePushA("#fcn#div_s32#" MW_AT_LOCATION);
  if (denominator == 0) {
    emlrtDivisionByZeroErrorR2012b(nullptr, emlrtRootTLSGlobal);
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if ((numerator < 0) != (denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  nvtxRangePop();
  return quotient;
}

real_T emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier *parentId)
{
  real_T y;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#d_emlrt_marshallIn#" MW_AT_LINE);
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  nvtxRangePop();
  return y;
}

void emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier *parentId,
                      char_T y[23])
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                      const char_T *identifier, char_T y[23])
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
  nvtxRangePop();
}

real_T emlrt_marshallIn(const mxArray *a__output_of_length_,
                        const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  y = emlrt_marshallIn(emlrtAlias(a__output_of_length_), &thisId);
  emlrtDestroyArray(&a__output_of_length_);
  nvtxRangePop();
  return y;
}

void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                    emxArray_creal32_T *gpu,
                                    boolean_T needsCopy)
{
  creal32_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxEnsureCapacity_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(creal32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(creal32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                  emxArray_int32_T *gpu, boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  int32_T *newData;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_int32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(int32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(int32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_real32_T(const emxArray_real32_T *cpu,
                                   emxArray_real32_T *gpu, boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  real32_T *newData;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxEnsureCapacity_real32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                 emxArray_real_T *gpu, boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_creal32_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxFree_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_int32_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxFree_real32_T(emxArray_real32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real32_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                    const emxArray_creal32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyCpuToGpu_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                  const emxArray_int32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(int32_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                   const emxArray_real32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyCpuToGpu_real32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(gpu->data, cpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                 hipMemcpyHostToDevice),
      __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                    emxArray_creal32_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyGpuToCpu_creal32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                   emxArray_real32_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA(
      "#loop#gpuEmxMemcpyGpuToCpu_real32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(real32_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyGpuToCpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_creal32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_creal32_T));
  nvtxRangePop();
}

void gpuEmxReset_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_int32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_int32_T));
  nvtxRangePop();
}

void gpuEmxReset_real32_T(emxArray_real32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real32_T));
  nvtxRangePop();
}

void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real_T));
  nvtxRangePop();
}

void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                    const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  nvtxRangePushA("#fcn#raiseCudaError#" MW_AT_LOCATION);
  len = strlen(file);
  pn = static_cast<char_T *>(std::calloc(static_cast<uint32_T>(len + 1UL), 1U));
  fn = static_cast<char_T *>(std::calloc(static_cast<uint32_T>(len + 1UL), 1U));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

real_T rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T a;
  real_T b;
  real_T y;
  nvtxRangePushA("#fcn#rt_hypotd_snf#" MW_AT_LOCATION);
  a = std::abs(u0);
  b = std::abs(u1);
  if (a < b) {
    a /= b;
    y = b * std::sqrt(a * a + 1.0);
  } else if (a > b) {
    b /= a;
    y = a * std::sqrt(b * b + 1.0);
  } else if (std::isnan(b)) {
    y = rtNaN;
  } else {
    y = a * 1.4142135623730951;
  }
  nvtxRangePop();
  return y;
}

real_T rt_powd_snf(real_T u0, real_T u1)
{
  real_T y;
  nvtxRangePushA("#fcn#rt_powd_snf#" MW_AT_LOCATION);
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else {
    real_T b;
    real_T c;
    b = std::abs(u0);
    c = std::abs(u1);
    if (std::isinf(u1)) {
      if (b == 1.0) {
        y = 1.0;
      } else if (b > 1.0) {
        if (u1 > 0.0) {
          y = rtInf;
        } else {
          y = 0.0;
        }
      } else if (u1 > 0.0) {
        y = 0.0;
      } else {
        y = rtInf;
      }
    } else if (c == 0.0) {
      y = 1.0;
    } else if (c == 1.0) {
      if (u1 > 0.0) {
        y = u0;
      } else {
        y = 1.0 / u0;
      }
    } else if (u1 == 2.0) {
      y = u0 * u0;
    } else if ((u1 == 0.5) && (u0 >= 0.0)) {
      y = std::sqrt(u0);
    } else if ((u0 < 0.0) && (u1 > std::floor(u1))) {
      y = rtNaN;
    } else {
      y = std::pow(u0, u1);
    }
  }
  nvtxRangePop();
  return y;
}

// End of code generation (ec_wt_fp_mexutil.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// firls.cu
//
// Code generation for function 'firls'
//

// Include files
#include "firls.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "mldivide.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cfloat>
#include <cmath>

// Variable Definitions
static emlrtRTEInfo df_emlrtRTEI{
    245,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ef_emlrtRTEI{
    243,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ff_emlrtRTEI{
    247,                                                     // lineNo
    5,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo gf_emlrtRTEI{
    258,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo hf_emlrtRTEI{
    274,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo if_emlrtRTEI{
    275,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo jf_emlrtRTEI{
    276,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo kf_emlrtRTEI{
    277,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo lf_emlrtRTEI{
    286,                                                     // lineNo
    5,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo mf_emlrtRTEI{
    281,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo nf_emlrtRTEI{
    341,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo of_emlrtRTEI{
    347,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo
    pf_emlrtRTEI{
        76,                  // lineNo
        9,                   // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo qf_emlrtRTEI{
    1,                                                               // lineNo
    21,                                                              // colNo
    "flipud",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/flipud.m" // pName
};

static emlrtRTEInfo rf_emlrtRTEI{
    341,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo sf_emlrtRTEI{
    353,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo tf_emlrtRTEI{
    355,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo uf_emlrtRTEI{
    253,                                                     // lineNo
    20,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo vf_emlrtRTEI{
    274,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo wf_emlrtRTEI{
    275,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo xf_emlrtRTEI{
    276,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo yf_emlrtRTEI{
    277,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo ag_emlrtRTEI{
    1,                                                       // lineNo
    16,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo bg_emlrtRTEI{
    295,                                                     // lineNo
    33,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo cg_emlrtRTEI{
    355,                                                     // lineNo
    17,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo dg_emlrtRTEI{
    297,                                                     // lineNo
    46,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo eg_emlrtRTEI{
    298,                                                     // lineNo
    39,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo rh_emlrtRTEI{
    297,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo sh_emlrtRTEI{
    297,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo th_emlrtRTEI{
    295,                                                     // lineNo
    13,                                                      // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

static emlrtRTEInfo uh_emlrtRTEI{
    295,                                                     // lineNo
    9,                                                       // colNo
    "firls",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/firls.m" // pName
};

// Function Declarations
static void binary_expand_op_8(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3, real_T in4,
                               const emxArray_real_T *in5);

static void binary_expand_op_9(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3,
                               const emxArray_real_T *in4,
                               const emxArray_real_T *in5);

static real_T rt_remd_snf(real_T u0, real_T u1);

// Function Definitions
static void binary_expand_op_8(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3, real_T in4,
                               const emxArray_real_T *in5)
{
  emxArray_real_T *b_in1;
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  nvtxRangePushA("#fcn#binary_expand_op_8#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_in1, 1, &rh_emlrtRTEI, true);
  i = b_in1->size[0];
  if (in3->size[0] == 1) {
    b_in1->size[0] = in1->size[0];
  } else {
    b_in1->size[0] = in3->size[0];
  }
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_in1, i, &rh_emlrtRTEI);
  stride_0_0 = (in1->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  if (in3->size[0] == 1) {
    b = in1->size[0];
  } else {
    b = in3->size[0];
  }
  profileLoopStart("binary_expand_op_8_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    b_in1->data[i] =
        in1->data[i * stride_0_0] +
        (in2 * in3->data[i * stride_1_0] - in4 * in5->data[i * stride_1_0]);
  }
  profileLoopEnd();
  i = in1->size[0];
  in1->size[0] = b_in1->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(in1, i, &sh_emlrtRTEI);
  profileLoopStart("binary_expand_op_8_loop_1", __LINE__,
                   (b_in1->size[0] - 1) + 1, "");
  for (i = 0; i < b_in1->size[0]; i++) {
    in1->data[i] = b_in1->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_in1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

static void binary_expand_op_9(emxArray_real_T *in1, real_T in2,
                               const emxArray_real_T *in3,
                               const emxArray_real_T *in4,
                               const emxArray_real_T *in5)
{
  emxArray_real_T *b_in1;
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  int32_T stride_2_0;
  nvtxRangePushA("#fcn#binary_expand_op_9#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_in1, 1, &th_emlrtRTEI, true);
  i = b_in1->size[0];
  if (in5->size[0] == 1) {
    b = in3->size[0];
  } else {
    b = in5->size[0];
  }
  if (b == 1) {
    b_in1->size[0] = in1->size[0];
  } else if (in5->size[0] == 1) {
    b_in1->size[0] = in3->size[0];
  } else {
    b_in1->size[0] = in5->size[0];
  }
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_in1, i, &th_emlrtRTEI);
  stride_0_0 = (in1->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  stride_2_0 = (in5->size[0] != 1);
  if (in5->size[0] == 1) {
    b = in3->size[0];
  } else {
    b = in5->size[0];
  }
  if (b == 1) {
    b = in1->size[0];
  } else if (in5->size[0] == 1) {
    b = in3->size[0];
  } else {
    b = in5->size[0];
  }
  profileLoopStart("binary_expand_op_9_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    b_in1->data[i] =
        in1->data[i * stride_0_0] +
        in2 * (in3->data[i * stride_1_0] - in4->data[i * stride_1_0]) /
            (in5->data[i * stride_2_0] * in5->data[i * stride_2_0]);
  }
  profileLoopEnd();
  i = in1->size[0];
  in1->size[0] = b_in1->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(in1, i, &uh_emlrtRTEI);
  profileLoopStart("binary_expand_op_9_loop_1", __LINE__,
                   (b_in1->size[0] - 1) + 1, "");
  for (i = 0; i < b_in1->size[0]; i++) {
    in1->data[i] = b_in1->data[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_in1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

static real_T rt_remd_snf(real_T u0, real_T u1)
{
  real_T y;
  nvtxRangePushA("#fcn#rt_remd_snf#" MW_AT_LOCATION);
  if (std::isnan(u0) || std::isnan(u1) || std::isinf(u0)) {
    y = rtNaN;
  } else if (std::isinf(u1)) {
    y = u0;
  } else if ((u1 != 0.0) && (u1 != std::trunc(u1))) {
    real_T q;
    q = std::abs(u0 / u1);
    if (!(std::abs(q - std::floor(q + 0.5)) > DBL_EPSILON * q)) {
      y = 0.0 * u0;
    } else {
      y = std::fmod(u0, u1);
    }
  } else {
    y = std::fmod(u0, u1);
  }
  nvtxRangePop();
  return y;
}

//
//
namespace coder {
void firls(real_T varargin_1, const real_T varargin_2[4], emxArray_real_T *h)
{
  static const int8_T A[4]{1, 1, 0, 0};
  emxArray_real_T gpu_a;
  emxArray_real_T *G;
  emxArray_real_T *b;
  emxArray_real_T *b0;
  emxArray_real_T *b_dv;
  emxArray_real_T *b_dv1;
  emxArray_real_T *b_y;
  emxArray_real_T *cpu_a;
  emxArray_real_T *k;
  emxArray_real_T *m;
  emxArray_real_T *sinc1A;
  emxArray_real_T *sinc2A;
  emxArray_real_T *sinc3A;
  emxArray_real_T *sinc4A;
  emxArray_real_T *x;
  emxArray_real_T *y;
  real_T f1;
  real_T max_freq;
  real_T min_freq;
  int32_T i;
  boolean_T Nodd;
  boolean_T a_outdatedOnCpu;
  boolean_T a_outdatedOnGpu;
  nvtxRangePushA("#fcn#firls#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_a);
  a_outdatedOnCpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  max_freq = 0.0;
  min_freq = 0.0;
  profileLoopStart("firls_loop_0", __LINE__, 2 + 1, "");
  for (i = 0; i < 3; i++) {
    f1 = varargin_2[i + 1];
    if (std::isnan(f1)) {
      Nodd = false;
    } else if (std::isnan(max_freq)) {
      Nodd = true;
    } else {
      Nodd = (max_freq < f1);
    }
    if (Nodd) {
      max_freq = f1;
    }
    if (std::isnan(f1)) {
      Nodd = false;
    } else if (std::isnan(min_freq)) {
      Nodd = true;
    } else {
      Nodd = (min_freq > f1);
    }
    if (Nodd) {
      min_freq = f1;
    }
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&m, 2, &ef_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&k, 1, &ff_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&G, 2, &uf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&sinc1A, 2, &vf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&sinc2A, 2, &wf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&sinc3A, 2, &xf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&sinc4A, 2, &yf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b, 1, &lf_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_a, 1, &ag_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_dv, 1, &bg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&x, 1, &cg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_dv1, 1, &bg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&y, 1, &dg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_y, 1, &eg_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b0, 1, &nf_emlrtRTEI, true);
  if ((!(max_freq > 1.0)) && (!(min_freq < 0.0))) {
    real_T F[4];
    real_T L;
    real_T b_b0;
    real_T tmpStorageLen;
    int32_T b_i;
    int32_T i1Start;
    int32_T loop_ub;
    int32_T md2;
    int32_T nG;
    int32_T nx;
    F[0] = varargin_2[0] / 2.0;
    F[1] = varargin_2[1] / 2.0;
    F[2] = varargin_2[2] / 2.0;
    F[3] = varargin_2[3] / 2.0;
    L = ((varargin_1 + 1.0) - 1.0) / 2.0;
    nvtxMarkA("#rt_remd_snf#" MW_AT_LINE);
    Nodd = (rt_remd_snf(varargin_1 + 1.0, 2.0) == 1.0);
    b_b0 = 0.0;
    if (!Nodd) {
      b_i = m->size[0] * m->size[1];
      m->size[0] = 1;
      m->size[1] = static_cast<int32_T>(L) + 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(m, b_i, &ef_emlrtRTEI);
      profileLoopStart("firls_loop_2", __LINE__, static_cast<int32_T>(L) + 1,
                       "");
      for (b_i = 0; b_i <= static_cast<int32_T>(L); b_i++) {
        m->data[b_i] = static_cast<real_T>(b_i) + 0.5;
      }
      profileLoopEnd();
    } else {
      b_i = m->size[0] * m->size[1];
      m->size[0] = 1;
      m->size[1] = static_cast<int32_T>(L) + 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(m, b_i, &df_emlrtRTEI);
      profileLoopStart("firls_loop_1", __LINE__, static_cast<int32_T>(L) + 1,
                       "");
      for (b_i = 0; b_i <= static_cast<int32_T>(L); b_i++) {
        m->data[b_i] = b_i;
      }
      profileLoopEnd();
    }
    b_i = k->size[0];
    k->size[0] = m->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(k, b_i, &ff_emlrtRTEI);
    profileLoopStart("firls_loop_3", __LINE__, (m->size[1] - 1) + 1, "");
    for (b_i = 0; b_i < m->size[1]; b_i++) {
      k->data[b_i] = m->data[b_i];
    }
    profileLoopEnd();
    a_outdatedOnGpu = (F[2] - F[1] != 0.0);
    nG = k->size[0];
    if (a_outdatedOnGpu) {
      b_i = G->size[0] * G->size[1];
      G->size[0] = k->size[0];
      G->size[1] = k->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(G, b_i, &gf_emlrtRTEI);
      profileLoopStart("firls_loop_4", __LINE__,
                       (k->size[0] * k->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < k->size[0] * k->size[0]; b_i++) {
        G->data[b_i] = 0.0;
      }
      profileLoopEnd();
      tmpStorageLen = 2.0 * static_cast<real_T>(k->size[0]) - 1.0;
    } else {
      G->size[0] = 0;
      G->size[1] = 0;
      tmpStorageLen = 0.0;
    }
    b_i = sinc1A->size[0] * sinc1A->size[1];
    sinc1A->size[0] = 1;
    sinc1A->size[1] = static_cast<int32_T>(tmpStorageLen);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(sinc1A, b_i, &hf_emlrtRTEI);
    b_i = sinc2A->size[0] * sinc2A->size[1];
    sinc2A->size[0] = 1;
    sinc2A->size[1] = static_cast<int32_T>(tmpStorageLen);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(sinc2A, b_i, &if_emlrtRTEI);
    b_i = sinc3A->size[0] * sinc3A->size[1];
    sinc3A->size[0] = 1;
    sinc3A->size[1] = static_cast<int32_T>(tmpStorageLen);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(sinc3A, b_i, &jf_emlrtRTEI);
    b_i = sinc4A->size[0] * sinc4A->size[1];
    sinc4A->size[0] = 1;
    sinc4A->size[1] = static_cast<int32_T>(tmpStorageLen);
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(sinc4A, b_i, &kf_emlrtRTEI);
    if (Nodd) {
      i1Start = -1;
      if (k->size[0] < 2) {
        i = 0;
        nx = 0;
      } else {
        i = 1;
        nx = k->size[0];
      }
      loop_ub = nx - i;
      profileLoopStart("firls_loop_6", __LINE__, (loop_ub - 1) + 1, "");
      for (b_i = 0; b_i < loop_ub; b_i++) {
        k->data[b_i] = k->data[i + b_i];
      }
      profileLoopEnd();
      b_i = k->size[0];
      k->size[0] = nx - i;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(k, b_i, &mf_emlrtRTEI);
    } else {
      i1Start = 0;
    }
    b_i = b->size[0];
    b->size[0] = k->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b, b_i, &lf_emlrtRTEI);
    profileLoopStart("firls_loop_5", __LINE__, (k->size[0] - 1) + 1, "");
    for (b_i = 0; b_i < k->size[0]; b_i++) {
      b->data[b_i] = 0.0;
    }
    profileLoopEnd();
    profileLoopStart("firls_loop_7", __LINE__, 1 + 1, "");
    for (int32_T s{0}; s < 2; s++) {
      real_T a;
      real_T b1;
      int8_T i1;
      i = s << 1;
      i1 = A[i];
      f1 = F[i + 1];
      max_freq = F[i];
      min_freq = static_cast<real_T>(A[i + 1] - i1) / (f1 - max_freq);
      b1 = static_cast<real_T>(i1) - min_freq * max_freq;
      if (Nodd) {
        b_b0 += b1 * (f1 - max_freq) +
                min_freq / 2.0 * (f1 * f1 - max_freq * max_freq);
      }
      b_i = b_dv->size[0];
      b_dv->size[0] = k->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_dv, b_i, &pf_emlrtRTEI);
      profileLoopStart("firls_loop_11", __LINE__, (k->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < k->size[0]; b_i++) {
        b_dv->data[b_i] = 6.2831853071795862 * k->data[b_i] * F[i + 1];
      }
      profileLoopEnd();
      nx = b_dv->size[0];
      b_i = b_dv1->size[0];
      b_dv1->size[0] = k->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_dv1, b_i, &pf_emlrtRTEI);
      profileLoopStart("firls_loop_14", __LINE__, (nx - 1) + 1, "");
      for (loop_ub = 0; loop_ub < nx; loop_ub++) {
        b_dv->data[loop_ub] = std::cos(b_dv->data[loop_ub]);
        b_dv1->data[loop_ub] = 6.2831853071795862 * k->data[loop_ub] * F[i];
      }
      profileLoopEnd();
      nx = b_dv1->size[0];
      profileLoopStart("firls_loop_16", __LINE__, (nx - 1) + 1, "");
      for (loop_ub = 0; loop_ub < nx; loop_ub++) {
        b_dv1->data[loop_ub] = std::cos(b_dv1->data[loop_ub]);
      }
      profileLoopEnd();
      a = min_freq / 39.478417604357432;
      if (b_dv->size[0] == 1) {
        b_i = k->size[0];
      } else {
        b_i = b_dv->size[0];
      }
      if ((b_dv->size[0] == k->size[0]) && (b->size[0] == b_i)) {
        profileLoopStart("firls_loop_20", __LINE__, (b->size[0] - 1) + 1, "");
        for (b_i = 0; b_i < b->size[0]; b_i++) {
          b->data[b_i] += a * (b_dv->data[b_i] - b_dv1->data[b_i]) /
                          (k->data[b_i] * k->data[b_i]);
        }
        profileLoopEnd();
      } else {
        nvtxMarkA("#binary_expand_op_9#" MW_AT_LINE);
        binary_expand_op_9(b, a, b_dv, b_dv1, k);
      }
      b_i = y->size[0];
      y->size[0] = k->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(y, b_i, &pf_emlrtRTEI);
      profileLoopStart("firls_loop_21", __LINE__, (k->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < k->size[0]; b_i++) {
        y->data[b_i] = 2.0 * k->data[b_i] * F[i + 1];
      }
      profileLoopEnd();
      f1 = F[i + 1];
      a = f1 * (min_freq * f1 + b1);
      md2 = y->size[0];
      b_i = b_y->size[0];
      b_y->size[0] = k->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_y, b_i, &pf_emlrtRTEI);
      profileLoopStart("firls_loop_22", __LINE__, (md2 - 1) + 1, "");
      for (loop_ub = 0; loop_ub < md2; loop_ub++) {
        if (std::abs(y->data[loop_ub]) < 1.0020841800044864E-292) {
          y->data[loop_ub] = 1.0;
        } else {
          f1 = 3.1415926535897931 * y->data[loop_ub];
          f1 = std::sin(f1) / f1;
          y->data[loop_ub] = f1;
        }
        b_y->data[loop_ub] = 2.0 * k->data[loop_ub] * F[i];
      }
      profileLoopEnd();
      f1 = F[i];
      max_freq = f1 * (min_freq * f1 + b1);
      nx = b_y->size[0];
      profileLoopStart("firls_loop_23", __LINE__, (nx - 1) + 1, "");
      for (loop_ub = 0; loop_ub < nx; loop_ub++) {
        if (std::abs(b_y->data[loop_ub]) < 1.0020841800044864E-292) {
          b_y->data[loop_ub] = 1.0;
        } else {
          f1 = 3.1415926535897931 * b_y->data[loop_ub];
          f1 = std::sin(f1) / f1;
          b_y->data[loop_ub] = f1;
        }
      }
      profileLoopEnd();
      if (b->size[0] == y->size[0]) {
        profileLoopStart("firls_loop_24", __LINE__, (b->size[0] - 1) + 1, "");
        for (b_i = 0; b_i < b->size[0]; b_i++) {
          b->data[b_i] += a * y->data[b_i] - max_freq * b_y->data[b_i];
        }
        profileLoopEnd();
      } else {
        nvtxMarkA("#binary_expand_op_8#" MW_AT_LINE);
        binary_expand_op_8(b, a, y, max_freq, b_y);
      }
      if (a_outdatedOnGpu) {
        real_T f2;
        uint32_T b_i1;
        f1 = 2.0 * F[i + 1];
        f2 = 2.0 * F[i];
        nx = static_cast<int32_T>(tmpStorageLen);
        profileLoopStart("firls_loop_25", __LINE__, (nx - 1) + 1, "");
        for (md2 = 0; md2 < nx; md2++) {
          b_i1 = (static_cast<uint32_T>(md2) + static_cast<uint32_T>(i1Start)) +
                 1U;
          max_freq = (static_cast<real_T>(md2) + 1.0) - static_cast<real_T>(nG);
          min_freq = f1 * static_cast<real_T>(b_i1);
          b1 = f2 * static_cast<real_T>(b_i1);
          a = f1 * max_freq;
          max_freq *= f2;
          if (std::abs(min_freq) < 1.0020841800044864E-292) {
            sinc1A->data[md2] = 1.0;
          } else {
            min_freq *= 3.1415926535897931;
            sinc1A->data[md2] = std::sin(min_freq) / min_freq;
          }
          if (std::abs(b1) < 1.0020841800044864E-292) {
            sinc2A->data[md2] = 1.0;
          } else {
            b1 *= 3.1415926535897931;
            sinc2A->data[md2] = std::sin(b1) / b1;
          }
          if (std::abs(a) < 1.0020841800044864E-292) {
            sinc3A->data[md2] = 1.0;
          } else {
            a *= 3.1415926535897931;
            sinc3A->data[md2] = std::sin(a) / a;
          }
          if (std::abs(max_freq) < 1.0020841800044864E-292) {
            sinc4A->data[md2] = 1.0;
          } else {
            max_freq *= 3.1415926535897931;
            sinc4A->data[md2] = std::sin(max_freq) / max_freq;
          }
        }
        profileLoopEnd();
        profileLoopStart("firls_loop_26", __LINE__, (nG - 1) + 1, "");
        for (md2 = 0; md2 < nG; md2++) {
          profileLoopStart("firls_loop_27", __LINE__, (nG - 1) + 1, "");
          for (nx = 0; nx < nG; nx++) {
            b_i1 =
                (static_cast<uint32_T>(nx) + static_cast<uint32_T>(md2)) + 1U;
            max_freq = static_cast<real_T>(nx - md2) + static_cast<real_T>(nG);
            G->data[nx + G->size[0] * md2] +=
                0.25 *
                (f1 * (sinc1A->data[static_cast<int32_T>(b_i1) - 1] +
                       sinc3A->data[static_cast<int32_T>(max_freq) - 1]) -
                 f2 * (sinc2A->data[static_cast<int32_T>(b_i1) - 1] +
                       sinc4A->data[static_cast<int32_T>(max_freq) - 1]));
          }
          profileLoopEnd();
        }
        profileLoopEnd();
      }
    }
    profileLoopEnd();
    if (Nodd) {
      b_i = b0->size[0];
      b0->size[0] = b->size[0] + 1;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b0, b_i, &nf_emlrtRTEI);
      b0->data[0] = b_b0;
      loop_ub = b->size[0];
      profileLoopStart("firls_loop_9", __LINE__, (loop_ub - 1) + 1, "");
      for (b_i = 0; b_i < loop_ub; b_i++) {
        b0->data[b_i + 1] = b->data[b_i];
      }
      profileLoopEnd();
      b_i = b->size[0];
      b->size[0] = b0->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b, b_i, &rf_emlrtRTEI);
      profileLoopStart("firls_loop_12", __LINE__, (b0->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < b0->size[0]; b_i++) {
        b->data[b_i] = b0->data[b_i];
      }
      profileLoopEnd();
    }
    if (a_outdatedOnGpu) {
      nvtxMarkA("#mldivide#" MW_AT_LINE);
      mldivide(G, b, cpu_a, &a_outdatedOnCpu, &gpu_a, &a_outdatedOnGpu);
    } else {
      b_i = cpu_a->size[0];
      cpu_a->size[0] = b->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_a, b_i, &of_emlrtRTEI);
      profileLoopStart("firls_loop_8", __LINE__, (b->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < b->size[0]; b_i++) {
        cpu_a->data[b_i] = 4.0 * b->data[b_i];
      }
      profileLoopEnd();
      if (Nodd) {
        cpu_a->data[0] /= 2.0;
      }
    }
    if (Nodd) {
      if (L + 1.0 < 2.0) {
        md2 = 0;
        nx = 1;
        i = -1;
        nG = 0;
        i1Start = 0;
      } else {
        md2 = static_cast<int32_T>(L + 1.0) - 1;
        nx = -1;
        i = 1;
        nG = 1;
        i1Start = static_cast<int32_T>(L + 1.0);
      }
      b_i = h->size[0] * h->size[1];
      h->size[0] = 1;
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      h->size[1] = ((div_s32(i - md2, nx) + i1Start) - nG) + 2;
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(h, b_i, &sf_emlrtRTEI);
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      loop_ub = div_s32(i - md2, nx);
      profileLoopStart("firls_loop_15", __LINE__, loop_ub + 1, "");
      for (b_i = 0; b_i <= loop_ub; b_i++) {
        if (a_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
        }
        a_outdatedOnCpu = false;
        h->data[b_i] = cpu_a->data[md2 + nx * b_i] / 2.0;
      }
      profileLoopEnd();
      if (a_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
      }
      nvtxMarkA("#div_s32#" MW_AT_LINE);
      h->data[div_s32(i - md2, nx) + 1] = cpu_a->data[0];
      loop_ub = i1Start - nG;
      profileLoopStart("firls_loop_18", __LINE__, (loop_ub - 1) + 1, "");
      for (b_i = 0; b_i < loop_ub; b_i++) {
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        h->data[(b_i + div_s32(i - md2, nx)) + 2] = cpu_a->data[nG + b_i] / 2.0;
      }
      profileLoopEnd();
    } else {
      b_i = x->size[0];
      x->size[0] = cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(x, b_i, &qf_emlrtRTEI);
      profileLoopStart("firls_loop_10", __LINE__, (cpu_a->size[0] - 1) + 1, "");
      for (b_i = 0; b_i < cpu_a->size[0]; b_i++) {
        if (a_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
        }
        a_outdatedOnCpu = false;
        x->data[b_i] = cpu_a->data[b_i];
      }
      profileLoopEnd();
      nx = cpu_a->size[0] - 1;
      md2 = cpu_a->size[0] >> 1;
      profileLoopStart("firls_loop_13", __LINE__, (md2 - 1) + 1, "");
      for (i = 0; i < md2; i++) {
        max_freq = x->data[i];
        x->data[i] = x->data[nx - i];
        x->data[nx - i] = max_freq;
      }
      profileLoopEnd();
      b_i = h->size[0] * h->size[1];
      h->size[0] = 1;
      h->size[1] = x->size[0] + cpu_a->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(h, b_i, &tf_emlrtRTEI);
      loop_ub = x->size[0];
      profileLoopStart("firls_loop_17", __LINE__, (loop_ub - 1) + 1, "");
      for (b_i = 0; b_i < loop_ub; b_i++) {
        h->data[b_i] = 0.5 * x->data[b_i];
      }
      profileLoopEnd();
      loop_ub = cpu_a->size[0];
      profileLoopStart("firls_loop_19", __LINE__, (loop_ub - 1) + 1, "");
      for (b_i = 0; b_i < loop_ub; b_i++) {
        if (a_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_a, &gpu_a);
        }
        a_outdatedOnCpu = false;
        h->data[b_i + x->size[0]] = 0.5 * cpu_a->data[b_i];
      }
      profileLoopEnd();
    }
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b0);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_dv1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_dv);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_a);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&sinc4A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&sinc3A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&sinc2A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&sinc1A);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&G);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&k);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&m);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_a);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (firls.cu)

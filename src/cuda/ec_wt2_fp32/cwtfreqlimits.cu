#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtfreqlimits.cu
//
// Code generation for function 'cwtfreqlimits'
//

// Include files
#include "cwtfreqlimits.h"
#include "ec_wt2_fp_mexutil.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
real_T getFreqFromCutoffMorse(real_T cutoff, real_T cf, real_T ga, real_T be)
{
  real_T omegac;
  real_T psihat_workspace_alpha;
  real_T psihat_workspace_anorm;
  nvtxRangePushA("#fcn#getFreqFromCutoffMorse#" MW_AT_LOCATION);
  psihat_workspace_anorm =
      2.0 * std::exp(be / ga * ((std::log(ga) - std::log(be)) + 1.0));
  psihat_workspace_alpha = 2.0 * cutoff;
  nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
  omegac = rt_powd_snf(750.0, 1.0 / ga);
  nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
  nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
  if (psihat_workspace_alpha - psihat_workspace_anorm * rt_powd_snf(cf, be) *
                                   std::exp(-rt_powd_snf(cf, ga)) >=
      0.0) {
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    if (!(psihat_workspace_alpha - psihat_workspace_anorm *
                                       rt_powd_snf(omegac, be) *
                                       std::exp(-rt_powd_snf(omegac, ga)) ==
          psihat_workspace_alpha - psihat_workspace_anorm *
                                       rt_powd_snf(cf, be) *
                                       std::exp(-rt_powd_snf(cf, ga)))) {
      omegac = cf;
    }
  } else {
    real_T a;
    real_T fa;
    real_T fb;
    a = cf;
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    fa = psihat_workspace_alpha - psihat_workspace_anorm * rt_powd_snf(cf, be) *
                                      std::exp(-rt_powd_snf(cf, ga));
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
    fb = psihat_workspace_alpha - psihat_workspace_anorm *
                                      rt_powd_snf(omegac, be) *
                                      std::exp(-rt_powd_snf(omegac, ga));
    if (fa == 0.0) {
      omegac = cf;
    } else if (!(fb == 0.0)) {
      real_T c;
      real_T d;
      real_T e;
      real_T fc;
      boolean_T exitg1;
      fc = fb;
      c = omegac;
      e = 0.0;
      d = 0.0;
      exitg1 = false;
      nvtxRangePushA("#loop#getFreqFromCutoffMorse_whileloop_0##" MW_AT_LINE);
      while ((!exitg1) && ((fb != 0.0) && (a != omegac))) {
        real_T m;
        real_T toler;
        if ((fb > 0.0) == (fc > 0.0)) {
          c = a;
          fc = fa;
          d = omegac - a;
          e = d;
        }
        if (std::abs(fc) < std::abs(fb)) {
          a = omegac;
          omegac = c;
          c = a;
          fa = fb;
          fb = fc;
          fc = fa;
        }
        m = 0.5 * (c - omegac);
        toler = 4.4408920985006262E-16 * std::fmax(std::abs(omegac), 1.0);
        if ((std::abs(m) <= toler) || (fb == 0.0)) {
          exitg1 = true;
        } else {
          if ((std::abs(e) < toler) || (std::abs(fa) <= std::abs(fb))) {
            d = m;
            e = m;
          } else {
            real_T q;
            real_T s;
            s = fb / fa;
            if (a == c) {
              fa = 2.0 * m * s;
              q = 1.0 - s;
            } else {
              real_T r;
              q = fa / fc;
              r = fb / fc;
              fa = s * (2.0 * m * q * (q - r) - (omegac - a) * (r - 1.0));
              q = (q - 1.0) * (r - 1.0) * (s - 1.0);
            }
            if (fa > 0.0) {
              q = -q;
            } else {
              fa = -fa;
            }
            if ((2.0 * fa < 3.0 * m * q - std::abs(toler * q)) &&
                (fa < std::abs(0.5 * e * q))) {
              e = d;
              d = fa / q;
            } else {
              d = m;
              e = m;
            }
          }
          a = omegac;
          fa = fb;
          if (std::abs(d) > toler) {
            omegac += d;
          } else if (omegac > c) {
            omegac -= toler;
          } else {
            omegac += toler;
          }
          nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
          nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
          fb = psihat_workspace_alpha - psihat_workspace_anorm *
                                            rt_powd_snf(omegac, be) *
                                            std::exp(-rt_powd_snf(omegac, ga));
        }
      }
      nvtxRangePop();
    }
  }
  nvtxRangePop();
  return omegac;
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (cwtfreqlimits.cu)

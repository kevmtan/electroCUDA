#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wt.cu
//
// Code generation for function 'wt'
//

// Include files
#include "wt.h"
#include "ec_wt2_fp_data.h"
#include "ec_wt2_fp_emxutil.h"
#include "ec_wt2_fp_mexutil.h"
#include "ec_wt2_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hipfft/hipfft.h"
#include "nvtx3/nvToolsExt.h"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo sc_emlrtRTEI{
    102,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    135,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    143,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "cuFFTNDCallback", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "cuFFTNDCallback.p" // pName
};

static emlrtRTEInfo wc_emlrtRTEI{
    138,  // lineNo
    19,   // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo xc_emlrtRTEI{
    138,  // lineNo
    6,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo yc_emlrtRTEI{
    161,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo ad_emlrtRTEI{
    180,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo bd_emlrtRTEI{
    151,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

// Function Declarations
static __global__ void cwtfilterbank_wt_kernel14(const int32_T i,
                                                 const int32_T b,
                                                 emxArray_real32_T x);

static __global__ void cwtfilterbank_wt_kernel15(const uint32_T xSize_dim1,
                                                 const int32_T b,
                                                 emxArray_creal32_T xposdft);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel14(
    const int32_T i, const int32_T b, emxArray_real32_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    int32_T j2;
    real32_T xtmp;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel15(
    const uint32_T xSize_dim1, const int32_T b, emxArray_creal32_T xposdft)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    b_j1 = static_cast<int32_T>(idx);
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].re =
        xposdft.data[b_j1 + 1].re;
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].im =
        -xposdft.data[b_j1 + 1].im;
  }
}

//
//
namespace coder {
void cwtfilterbank_wt(const cwtfilterbank *self, const emxArray_real32_T *x,
                      emxArray_creal32_T *varargout_1)
{
  dim3 block;
  dim3 grid;
  emxArray_creal32_T gpu_cfsposdft;
  emxArray_creal32_T gpu_xposdft;
  emxArray_creal32_T *cpu_cfsposdft;
  emxArray_creal32_T *cpu_xposdft;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_xv;
  emxArray_real32_T *cpu_x;
  emxArray_real32_T *cpu_xv;
  emxArray_real32_T *psihat;
  real_T d;
  int32_T acoef;
  int32_T bcoef;
  int32_T csz_idx_1;
  int32_T i;
  int32_T i2;
  int32_T i3;
  int32_T nd2;
  uint32_T xSize[2];
  boolean_T validLaunchParams;
  boolean_T x_outdatedOnCpu;
  boolean_T xv_outdatedOnGpu;
  nvtxRangePushA("#fcn#cwtfilterbank_wt#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_cfsposdft);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_x);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_xv);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&psihat, 2, &sc_emlrtRTEI, true);
  bcoef = psihat->size[0] * psihat->size[1];
  psihat->size[0] = self->PsiDFT->size[0];
  psihat->size[1] = self->PsiDFT->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(psihat, bcoef, &sc_emlrtRTEI);
  profileLoopStart("cwtfilterbank_wt_loop_0", __LINE__,
                   (self->PsiDFT->size[0] * self->PsiDFT->size[1] - 1) + 1, "");
  for (bcoef = 0; bcoef < self->PsiDFT->size[0] * self->PsiDFT->size[1];
       bcoef++) {
    psihat->data[bcoef] = static_cast<real32_T>(self->PsiDFT->data[bcoef]);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_xv, 2, &tc_emlrtRTEI, true);
  xv_outdatedOnGpu = false;
  bcoef = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = x->size[0];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(cpu_xv, bcoef, &tc_emlrtRTEI);
  profileLoopStart("cwtfilterbank_wt_loop_1", __LINE__, (x->size[0] - 1) + 1,
                   "");
  for (bcoef = 0; bcoef < x->size[0]; bcoef++) {
    cpu_xv->data[bcoef] = x->data[bcoef];
    xv_outdatedOnGpu = true;
  }
  profileLoopEnd();
  if (self->SignalPad > 0.0) {
    d = self->SignalPad;
    if (d < 1.0) {
      i = 0;
    } else {
      i = static_cast<int32_T>(d);
    }
    d = (static_cast<real_T>(x->size[0]) - self->SignalPad) + 1.0;
    if (d > x->size[0]) {
      csz_idx_1 = 0;
      i2 = 1;
      i3 = -1;
    } else {
      csz_idx_1 = x->size[0] - 1;
      i2 = -1;
      i3 = static_cast<int32_T>(d) - 1;
    }
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&cpu_x, 2, &wc_emlrtRTEI, true);
    xv_outdatedOnGpu = false;
    bcoef = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = 1;
    cpu_x->size[1] = i;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_x, bcoef, &wc_emlrtRTEI);
    profileLoopStart("cwtfilterbank_wt_loop_3", __LINE__, (i - 1) + 1, "");
    for (bcoef = 0; bcoef < i; bcoef++) {
      cpu_x->data[bcoef] = x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters((i >> 1) - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !xv_outdatedOnGpu);
    if (xv_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwtfilterbank_wt_kernel14#" MW_AT_LINE);
      cwtfilterbank_wt_kernel14<<<grid, block>>>(i, (i >> 1) - 1, gpu_x);
    }
    x_outdatedOnCpu = true;
    xv_outdatedOnGpu = false;
    bcoef = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    cpu_xv->size[1] =
        ((x->size[0] + cpu_x->size[1]) + div_s32(i3 - csz_idx_1, i2)) + 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_xv, bcoef, &xc_emlrtRTEI);
    acoef = cpu_x->size[1];
    profileLoopStart("cwtfilterbank_wt_loop_5", __LINE__, (acoef - 1) + 1, "");
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      if (x_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_x, &gpu_x);
      }
      x_outdatedOnCpu = false;
      cpu_xv->data[bcoef] = cpu_x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    acoef = x->size[0];
    profileLoopStart("cwtfilterbank_wt_loop_7", __LINE__, (acoef - 1) + 1, "");
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      cpu_xv->data[bcoef + cpu_x->size[1]] = x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    acoef = div_s32(i3 - csz_idx_1, i2);
    profileLoopStart("cwtfilterbank_wt_loop_8", __LINE__, acoef + 1, "");
    for (bcoef = 0; bcoef <= acoef; bcoef++) {
      cpu_xv->data[(bcoef + x->size[0]) + cpu_x->size[1]] =
          x->data[csz_idx_1 + i2 * bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&cpu_x);
  }
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_xposdft, 2, &uc_emlrtRTEI, true);
  if (cpu_xv->size[1] == 0) {
    xv_outdatedOnGpu = false;
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = 0;
  } else if (cpu_xv->size[1] - (static_cast<int32_T>(
                                    static_cast<uint32_T>(cpu_xv->size[1]) >> 1)
                                << 1) ==
             1) {
    hipfftHandle b_fftPlanHandle;
    xv_outdatedOnGpu = false;
    bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = cpu_xv->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_xposdft, bcoef, &uc_emlrtRTEI);
    profileLoopStart("cwtfilterbank_wt_loop_2", __LINE__,
                     (cpu_xv->size[1] - 1) + 1, "");
    for (bcoef = 0; bcoef < cpu_xv->size[1]; bcoef++) {
      cpu_xposdft->data[bcoef].re = cpu_xv->data[bcoef];
      cpu_xposdft->data[bcoef].im = 0.0F;
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nd2 = cpu_xposdft->size[1];
    b_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_C2C, 1);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_xposdft, &gpu_xposdft,
                                   !xv_outdatedOnGpu);
    if (xv_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_xposdft, cpu_xposdft);
    }
    hipfftExecC2C(b_fftPlanHandle, (hipfftComplex *)&gpu_xposdft.data[0],
                 (hipfftComplex *)&gpu_xposdft.data[0], HIPFFT_FORWARD);
    xv_outdatedOnGpu = true;
  } else {
    hipfftHandle fftPlanHandle;
    nd2 = cpu_xv->size[1];
    xSize[1] = static_cast<uint32_T>(cpu_xv->size[1]);
    bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = cpu_xv->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_xposdft, bcoef, &vc_emlrtRTEI);
    fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_R2C, 1);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_xv, &gpu_xv, !xv_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_xposdft, &gpu_xposdft, true);
    if (xv_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_xv, cpu_xv);
    }
    hipfftExecR2C(fftPlanHandle, (hipfftReal *)&gpu_xv.data[0],
                 (hipfftComplex *)&gpu_xposdft.data[0]);
    nd2 = static_cast<int32_T>(static_cast<real_T>(cpu_xv->size[1] + 1) / 2.0);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nd2 - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwtfilterbank_wt_kernel15#" MW_AT_LINE);
      cwtfilterbank_wt_kernel15<<<grid, block>>>(xSize[1], nd2 - 2,
                                                 gpu_xposdft);
    }
    xv_outdatedOnGpu = true;
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_xv);
  nd2 = psihat->size[1];
  i = cpu_xposdft->size[1];
  if (nd2 <= i) {
    i = nd2;
  }
  if (psihat->size[1] == 1) {
    csz_idx_1 = cpu_xposdft->size[1];
  } else if (cpu_xposdft->size[1] == 1) {
    csz_idx_1 = psihat->size[1];
  } else if (cpu_xposdft->size[1] == psihat->size[1]) {
    csz_idx_1 = cpu_xposdft->size[1];
  } else {
    csz_idx_1 = i;
  }
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_cfsposdft, 2, &bd_emlrtRTEI, true);
  x_outdatedOnCpu = false;
  bcoef = cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
  cpu_cfsposdft->size[0] = psihat->size[0];
  nd2 = psihat->size[1];
  i = cpu_xposdft->size[1];
  if (nd2 <= i) {
    i = nd2;
  }
  if (psihat->size[1] == 1) {
    cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
  } else if (cpu_xposdft->size[1] == 1) {
    cpu_cfsposdft->size[1] = psihat->size[1];
  } else if (cpu_xposdft->size[1] == psihat->size[1]) {
    cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
  } else {
    cpu_cfsposdft->size[1] = i;
  }
  nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
  emxEnsureCapacity_creal32_T(cpu_cfsposdft, bcoef, &rc_emlrtRTEI);
  if ((psihat->size[0] != 0) && (csz_idx_1 != 0)) {
    acoef = (cpu_xposdft->size[1] != 1);
    bcoef = (psihat->size[1] != 1);
    csz_idx_1--;
    nd2 = (psihat->size[0] != 1);
    profileLoopStart("cwtfilterbank_wt_loop_4", __LINE__, csz_idx_1 + 1, "");
    for (int32_T k{0}; k <= csz_idx_1; k++) {
      i = acoef * k;
      i2 = bcoef * k;
      i3 = cpu_cfsposdft->size[0] - 1;
      profileLoopStart("cwtfilterbank_wt_loop_6", __LINE__, i3 + 1, "");
      for (int32_T b_k{0}; b_k <= i3; b_k++) {
        if (xv_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xposdft, &gpu_xposdft);
        }
        cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].re =
            psihat->data[nd2 * b_k + psihat->size[0] * i2] *
            cpu_xposdft->data[i].re;
        xv_outdatedOnGpu = false;
        cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].im =
            psihat->data[nd2 * b_k + psihat->size[0] * i2] *
            cpu_xposdft->data[i].im;
        x_outdatedOnCpu = true;
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_xposdft);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&psihat);
  if ((cpu_cfsposdft->size[0] == 0) || (cpu_cfsposdft->size[1] == 0)) {
    xSize[0] = static_cast<uint32_T>(cpu_cfsposdft->size[0]);
    xSize[1] = static_cast<uint32_T>(cpu_cfsposdft->size[1]);
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[0] = cpu_cfsposdft->size[0];
    varargout_1->size[1] = cpu_cfsposdft->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(varargout_1, bcoef, &yc_emlrtRTEI);
    if (static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1 >=
        0) {
      std::memset(&varargout_1->data[0], 0,
                  static_cast<uint32_T>(static_cast<int32_T>(xSize[0]) *
                                        static_cast<int32_T>(xSize[1])) *
                      sizeof(creal32_T));
    }
  } else {
    hipfftHandle c_fftPlanHandle;
    real32_T cfsposdft_re;
    nd2 = cpu_cfsposdft->size[1];
    c_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, cpu_cfsposdft->size[0], 1,
                                       HIPFFT_C2C, cpu_cfsposdft->size[0]);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_cfsposdft, &gpu_cfsposdft,
                                   !x_outdatedOnCpu);
    if (x_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_cfsposdft, cpu_cfsposdft);
    }
    hipfftExecC2C(c_fftPlanHandle, (hipfftComplex *)&gpu_cfsposdft.data[0],
                 (hipfftComplex *)&gpu_cfsposdft.data[0], HIPFFT_BACKWARD);
    xv_outdatedOnGpu = true;
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[0] = cpu_cfsposdft->size[0];
    varargout_1->size[1] = cpu_cfsposdft->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(varargout_1, bcoef, &yc_emlrtRTEI);
    cfsposdft_re = static_cast<real32_T>(cpu_cfsposdft->size[1]);
    profileLoopStart("cwtfilterbank_wt_loop_9", __LINE__,
                     (cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1] - 1) + 1,
                     "");
    for (bcoef = 0; bcoef < cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
         bcoef++) {
      real32_T ai;
      real32_T ar;
      if (xv_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_creal32_T(cpu_cfsposdft, &gpu_cfsposdft);
      }
      ar = cpu_cfsposdft->data[bcoef].re;
      xv_outdatedOnGpu = false;
      ai = cpu_cfsposdft->data[bcoef].im;
      if (ai == 0.0F) {
        varargout_1->data[bcoef].re = ar / cfsposdft_re;
        varargout_1->data[bcoef].im = 0.0F;
      } else if (ar == 0.0F) {
        varargout_1->data[bcoef].re = 0.0F;
        varargout_1->data[bcoef].im = ai / cfsposdft_re;
      } else {
        varargout_1->data[bcoef].re = ar / cfsposdft_re;
        varargout_1->data[bcoef].im = ai / cfsposdft_re;
      }
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_cfsposdft);
  if (self->SignalPad > 0.0) {
    real_T d3;
    d = self->SignalPad + 1.0;
    d3 = self->SignalPad + self->SignalLength;
    if (d > d3) {
      nd2 = 0;
      i = 0;
    } else {
      nd2 = static_cast<int32_T>(d) - 1;
      i = static_cast<int32_T>(d3);
    }
    csz_idx_1 = varargout_1->size[0];
    acoef = i - nd2;
    profileLoopStart("cwtfilterbank_wt_loop_10", __LINE__, (acoef - 1) + 1, "");
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      profileLoopStart("cwtfilterbank_wt_loop_11", __LINE__,
                       (csz_idx_1 - 1) + 1, "");
      for (i2 = 0; i2 < csz_idx_1; i2++) {
        varargout_1->data[i2 + csz_idx_1 * bcoef] =
            varargout_1->data[i2 + varargout_1->size[0] * (nd2 + bcoef)];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[1] = i - nd2;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(varargout_1, bcoef, &ad_emlrtRTEI);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_xv);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_cfsposdft);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (wt.cu)

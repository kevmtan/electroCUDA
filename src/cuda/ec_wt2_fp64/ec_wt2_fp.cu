#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt2_fp.cu
//
// Code generation for function 'ec_wt2_fp'
//

// Include files
#include "ec_wt2_fp.h"
#include "circshift.h"
#include "cwtfilterbank.h"
#include "ec_wt2_fp_data.h"
#include "ec_wt2_fp_emxutil.h"
#include "ec_wt2_fp_mexutil.h"
#include "ec_wt2_fp_types.h"
#include "rt_nonfinite.h"
#include "wt.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Variable Definitions
static emlrtMCInfo c_emlrtMCI{
    53,        // lineNo
    19,        // colNo
    "flt2str", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/coder/coder/lib/+coder/+internal/"
    "flt2str.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    23,                                                      // lineNo
    1,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    27,                                                      // lineNo
    27,                                                      // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    35,                                                      // lineNo
    28,                                                      // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    33,                                                      // lineNo
    29,                                                      // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    46,                                                      // lineNo
    1,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    58,                                                      // lineNo
    10,                                                      // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    35,                                                      // lineNo
    9,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    66,                          // lineNo
    30,                          // colNo
    "applyBinaryScalarFunction", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "applyBinaryScalarFunction.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    33,                                                      // lineNo
    9,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    91,                                                           // lineNo
    14,                                                           // colNo
    "downsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/downsample.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    50,                                                      // lineNo
    5,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    67,                                                      // lineNo
    5,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    21,                                                      // lineNo
    1,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    33,                                                      // lineNo
    17,                                                      // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    35,                                                      // lineNo
    17,                                                      // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    46,                                                      // lineNo
    6,                                                       // colNo
    "ec_wt2_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt2_fp.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23]);

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location);

static uint64_T computeNumIters(int32_T ub, int32_T b_ub);

static
#ifdef __HIPCC__
    __device__
#endif
        int32_T
        div_s32_device(int32_T numerator, int32_T denominator);

static __global__ void ec_wt2_fp_kernel1(const int32_T k,
                                         const emxArray_real_T x,
                                         const int32_T b_x, emxArray_real_T c_x,
                                         int32_T x_dim0);

static __global__ void ec_wt2_fp_kernel10(const emxArray_creal_T r,
                                          const int32_T b_r, const int32_T c_r,
                                          emxArray_creal_T yc, int32_T yc_dim0,
                                          int32_T r_dim0);

static __global__ void ec_wt2_fp_kernel11(const int32_T i1, const int32_T nx,
                                          const int32_T dim, int32_T sizeX[2]);

static __global__ void
ec_wt2_fp_kernel12(const emxArray_creal_T yc, const int32_T lshift_dim0,
                   const int32_T i1, const int32_T b, const int32_T c,
                   emxArray_creal_T b_yc, int32_T yc_dim0);

static __global__ void ec_wt2_fp_kernel13(const emxArray_creal_T yc,
                                          const int32_T lshift,
                                          emxArray_creal_T b_yc);

static __global__ void ec_wt2_fp_kernel2(const emxArray_creal_T r1,
                                         const int32_T b_r1, const int32_T c_r1,
                                         emxArray_creal_T x, int32_T x_dim0,
                                         int32_T r1_dim0);

static __global__ void ec_wt2_fp_kernel3(const emxArray_creal_T x,
                                         const int32_T b, emxArray_real_T yc);

static __global__ void ec_wt2_fp_kernel4(const emxArray_real_T yc,
                                         const int32_T b_yc, emxArray_real_T x);

static __global__ void ec_wt2_fp_kernel5(const emxArray_real_T x,
                                         const int32_T b, emxArray_real_T yc);

static __global__ void ec_wt2_fp_kernel6(const int32_T i1, const int32_T nx,
                                         const int32_T dim, int32_T sizeX[2]);

static __global__ void ec_wt2_fp_kernel7(const emxArray_real_T yc,
                                         const int32_T lshift_dim0,
                                         const int32_T i1, const int32_T b,
                                         const int32_T c, emxArray_real_T b_yc,
                                         int32_T yc_dim0);

static __global__ void ec_wt2_fp_kernel8(const emxArray_real_T yc,
                                         const int32_T lshift,
                                         emxArray_real_T b_yc);

static __global__ void ec_wt2_fp_kernel9(const int32_T k,
                                         const emxArray_real_T x,
                                         const int32_T b_x, emxArray_real_T c_x,
                                         int32_T x_dim0);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23]);

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23]);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23])
{
  static const int32_T dims[2]{1, 23};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 23);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  const mxArray *m3;
  nvtxRangePushA("#fcn#b_sprintf#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  m3 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                             "sprintf", true, location);
  nvtxRangePop();
  return m3;
}

static uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
  }
  numIters *= n;
  nvtxRangePop();
  return numIters;
}

static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if (static_cast<int32_T>(numerator < 0) !=
        static_cast<int32_T>(denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel1(
    const int32_T k, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T c_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_x.data[i] = x.data[i + x_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel10(
    const emxArray_creal_T r, const int32_T b_r, const int32_T c_r,
    emxArray_creal_T yc, int32_T yc_dim0, int32_T r_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_r) + 1UL) * (static_cast<uint64_T>(b_r) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r) + 1UL));
    yc.data[k + yc_dim0 * i].re = r.data[i + r_dim0 * k].re;
    yc.data[k + yc_dim0 * i].im = -r.data[i + r_dim0 * k].im;
  }
}

static __global__ __launch_bounds__(32, 1) void ec_wt2_fp_kernel11(
    const int32_T i1, const int32_T nx, const int32_T dim, int32_T sizeX[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    sizeX[dim] = div_s32_device(nx, i1) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel12(
    const emxArray_creal_T yc, const int32_T lshift_dim0, const int32_T i1,
    const int32_T b, const int32_T c, emxArray_creal_T b_yc, int32_T yc_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b) + 1UL));
    b_yc.data[k + yc_dim0 * i] = yc.data[i1 * k + lshift_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel13(
    const emxArray_creal_T yc, const int32_T lshift, emxArray_creal_T b_yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(lshift);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_yc.data[i] = yc.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel2(
    const emxArray_creal_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_creal_T x, int32_T x_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r1) + 1UL));
    x.data[k + x_dim0 * i].re = r1.data[i + r1_dim0 * k].re;
    x.data[k + x_dim0 * i].im = -r1.data[i + r1_dim0 * k].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel3(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    yc.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel4(
    const emxArray_real_T yc, const int32_T b_yc, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_yc);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = yc.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel5(
    const emxArray_real_T x, const int32_T b, emxArray_real_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    yc.data[k] = x.data[k] * x.data[k];
  }
}

static __global__ __launch_bounds__(32, 1) void ec_wt2_fp_kernel6(
    const int32_T i1, const int32_T nx, const int32_T dim, int32_T sizeX[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    sizeX[dim] = div_s32_device(nx, i1) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel7(
    const emxArray_real_T yc, const int32_T lshift_dim0, const int32_T i1,
    const int32_T b, const int32_T c, emxArray_real_T b_yc, int32_T yc_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b) + 1UL));
    b_yc.data[k + yc_dim0 * i] = yc.data[i1 * k + lshift_dim0 * i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel8(
    const emxArray_real_T yc, const int32_T lshift, emxArray_real_T b_yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(lshift);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    b_yc.data[i] = yc.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt2_fp_kernel9(
    const int32_T k, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T c_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_x.data[i] = x.data[i + x_dim0 * k];
  }
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23])
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23])
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
  nvtxRangePop();
}

//
// function [y,frqs] = ec_wt2_fp(x,fs,lims,voices,ds,doReal,doPwr)
void ec_wt2_fp(const emxArray_real_T *cpu_x, real_T fs, const real_T lims[2],
               real_T voices, real_T ds, boolean_T doReal, boolean_T doPwr,
               emxArray_cell_wrap_0 *y, emxArray_real_T *frqs)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  coder::cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal_T c_gpu_x;
  emxArray_creal_T c_gpu_yc;
  emxArray_creal_T d_gpu_yc;
  emxArray_creal_T gpu_r;
  emxArray_creal_T gpu_r1;
  emxArray_creal_T *b_cpu_x;
  emxArray_creal_T *b_cpu_yc;
  emxArray_creal_T *c_cpu_yc;
  emxArray_creal_T *cpu_r;
  emxArray_creal_T *cpu_r1;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_yc;
  emxArray_real_T d_gpu_x;
  emxArray_real_T e_gpu_x;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_yc;
  emxArray_real_T *c_cpu_x;
  emxArray_real_T *cpu_yc;
  emxArray_real_T *d_cpu_x;
  emxArray_real_T *d_cpu_yc;
  emxArray_real_T *e_cpu_x;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  int32_T(*gpu_lshift)[2];
  int32_T(*gpu_sizeX)[2];
  int32_T dim;
  int32_T i;
  int32_T i1;
  int32_T nx;
  int32_T vstride;
  char_T unusedExpr[23];
  boolean_T lshift_outdatedOnCpu;
  boolean_T lshift_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_wt2_fp#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_lshift, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_sizeX, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_yc);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&d_gpu_yc);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&e_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r1);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&c_gpu_yc);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_yc);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x);
  x_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInitStruct_cwtfilterbank#" MW_AT_LINE);
  emxInitStruct_cwtfilterbank(&fb, &o_emlrtRTEI, true);
  //  [electroCUDA] CWT - CUDA mex source
  //    Called by function 'ec_wt' (see for details)
  //    Intended to be compiled into a CUDA mex binary
  //    Kevin Tan, 2024 (github.com/kevmtan/electroCUDA)
  // 'ec_wt2_fp:7' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_wt2_fp:8' fs (1,1) double
  //  Sampling rate
  // 'ec_wt2_fp:9' lims (1,2) double
  //  Frequency limits
  // 'ec_wt2_fp:10' voices (1,1) double = 10
  //  Voices per octave
  // 'ec_wt2_fp:11' ds (1,1) double = 0
  //  Downsampling factor
  // 'ec_wt2_fp:12' doReal (1,1) logical = true
  //  Real output? (complex otherwise)
  // 'ec_wt2_fp:13' doPwr (1,1) logical = false
  //  Power output? (magnitude otherwise)
  // 'ec_wt2_fp:15' if ds<=1
  if (ds <= 1.0) {
    // 'ec_wt2_fp:15' ;
    // 'ec_wt2_fp:15' ds=0;
    ds = 0.0;
  }
  //  Prep
  // 'ec_wt2_fp:18' nChs = width(x);
  //  Generate CWT filter
  // 'ec_wt2_fp:21' fb =
  // cwtfilterbank(Wavelet="Morse",SignalLength=height(x),SamplingFrequency=fs,...
  // 'ec_wt2_fp:22'     FrequencyLimits=lims,VoicesPerOctave=voices);
  nvtxMarkA("#cwtfilterbank_cwtfilterbank#" MW_AT_LINE);
  coder::cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(cpu_x->size[0]),
                                     fs, lims, voices);
  // 'ec_wt2_fp:23' frqs = flip(fb.centerFrequencies);
  vstride = frqs->size[0];
  frqs->size[0] = fb.WaveletCenterFrequencies->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(frqs, vstride, &emlrtRTEI);
  profileLoopStart("ec_wt2_fp_loop_0", __LINE__,
                   (fb.WaveletCenterFrequencies->size[0] - 1) + 1, "");
  for (vstride = 0; vstride < fb.WaveletCenterFrequencies->size[0]; vstride++) {
    frqs->data[vstride] = fb.WaveletCenterFrequencies->data[vstride];
  }
  profileLoopEnd();
  dim = 2;
  if (fb.WaveletCenterFrequencies->size[0] != 1) {
    dim = 1;
  }
  if (fb.WaveletCenterFrequencies->size[0] != 0) {
    if (dim <= 1) {
      nx = fb.WaveletCenterFrequencies->size[0];
    } else {
      nx = 1;
    }
    if (nx > 1) {
      vstride = 1;
      profileLoopStart("ec_wt2_fp_loop_1", __LINE__, (dim - 2) + 1, "");
      for (int32_T k{0}; k <= dim - 2; k++) {
        vstride *= fb.WaveletCenterFrequencies->size[0];
      }
      profileLoopEnd();
      if (dim <= 1) {
        nx = fb.WaveletCenterFrequencies->size[0];
      } else {
        nx = 1;
      }
      dim = nx >> 1;
      i1 = vstride - 1;
      profileLoopStart("ec_wt2_fp_loop_2", __LINE__, i1 + 1, "");
      for (i = 0; i <= i1; i++) {
        profileLoopStart("ec_wt2_fp_loop_3", __LINE__, (dim - 1) + 1, "");
        for (int32_T k{0}; k < dim; k++) {
          real_T tmp;
          tmp = frqs->data[i + k * vstride];
          frqs->data[i + k * vstride] =
              frqs->data[i + ((nx - k) - 1) * vstride];
          frqs->data[i + ((nx - k) - 1) * vstride] = tmp;
        }
        profileLoopEnd();
      }
      profileLoopEnd();
    }
  }
  //  Preallocate output
  // 'ec_wt2_fp:26' coder.gpu.kernelfun;
  //  Trigger CUDA kernel creation
  // 'ec_wt2_fp:27' y = coder.nullcopy(cell(1,nChs));
  vstride = y->size[0] * y->size[1];
  y->size[0] = 1;
  y->size[1] = cpu_x->size[1];
  nvtxMarkA("#emxEnsureCapacity_cell_wrap_0#" MW_AT_LINE);
  emxEnsureCapacity_cell_wrap_0(y, vstride, &b_emlrtRTEI);
  //  Processing loop across channels
  // 'ec_wt2_fp:31' for ch = 1:nChs
  i = cpu_x->size[1];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_yc, 2, &p_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_yc, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_x, 2, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_x, 2, &i_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r, 2, &r_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r1, 2, &f_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_x, 1, &c_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&e_cpu_x, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&c_cpu_yc, 2, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_yc, 2, &k_emlrtRTEI, true);
  profileLoopStart("ec_wt2_fp_loop_4", __LINE__, (i - 1) + 1, "");
  for (int32_T k{0}; k < i; k++) {
    // 'ec_wt2_fp:32' if doReal
    if (doReal) {
      uint32_T b_dv[2];
      boolean_T validLaunchParams;
      boolean_T yc_outdatedOnCpu;
      // 'ec_wt2_fp:33' y{ch} = cwtR_lfn(fb,x(:,ch),ds,doPwr);
      //  Run CWT (magnitude/power)
      //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%% CWT - magnitude out
      // 'ec_wt2_fp:58' yc = abs(wt(fb,xc)');
      vstride = e_cpu_x->size[0];
      e_cpu_x->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(e_cpu_x, vstride, &d_emlrtRTEI);
      dim = cpu_x->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(dim), &grid, &block, 2147483647U);
      if (x_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
      }
      x_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(e_cpu_x, &b_gpu_x, true);
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
      }
      x_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt2_fp_kernel1#" MW_AT_LINE);
        ec_wt2_fp_kernel1<<<grid, block>>>(k, gpu_x, dim, b_gpu_x,
                                           cpu_x->size[0U]);
      }
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(e_cpu_x, &b_gpu_x);
      nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
      coder::cwtfilterbank_wt(&fb, e_cpu_x, cpu_r1);
      vstride = b_cpu_x->size[0] * b_cpu_x->size[1];
      b_cpu_x->size[0] = cpu_r1->size[1];
      b_cpu_x->size[1] = cpu_r1->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(b_cpu_x, vstride, &f_emlrtRTEI);
      dim = cpu_r1->size[0] - 1;
      nx = cpu_r1->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(dim, nx), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_r1, &gpu_r1, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(b_cpu_x, &c_gpu_x, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r1, cpu_r1);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt2_fp_kernel2#" MW_AT_LINE);
        ec_wt2_fp_kernel2<<<grid, block>>>(gpu_r1, nx, dim, c_gpu_x,
                                           b_cpu_x->size[0U], cpu_r1->size[0U]);
      }
      nx = b_cpu_x->size[0] * b_cpu_x->size[1];
      profileLoopStart("ec_wt2_fp_loop_5", __LINE__, 1 + 1, "");
      for (i1 = 0; i1 < 2; i1++) {
        b_dv[i1] = static_cast<uint32_T>(b_cpu_x->size[i1]);
      }
      profileLoopEnd();
      vstride = cpu_yc->size[0] * cpu_yc->size[1];
      cpu_yc->size[0] = static_cast<int32_T>(b_dv[0]);
      cpu_yc->size[1] = static_cast<int32_T>(b_dv[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_yc, vstride, &h_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_yc, &gpu_yc, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt2_fp_kernel3#" MW_AT_LINE);
        ec_wt2_fp_kernel3<<<grid, block>>>(c_gpu_x, nx - 1, gpu_yc);
      }
      yc_outdatedOnCpu = true;
      //  Convert to power
      // 'ec_wt2_fp:61' if doPwr
      if (doPwr) {
        // 'ec_wt2_fp:62' yc = yc.^2;
        vstride = c_cpu_x->size[0] * c_cpu_x->size[1];
        c_cpu_x->size[0] = cpu_yc->size[0];
        c_cpu_x->size[1] = cpu_yc->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_cpu_x, vstride, &i_emlrtRTEI);
        dim = cpu_yc->size[0] * cpu_yc->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(dim), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(c_cpu_x, &d_gpu_x, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt2_fp_kernel4#" MW_AT_LINE);
          ec_wt2_fp_kernel4<<<grid, block>>>(gpu_yc, dim, d_gpu_x);
        }
        profileLoopStart("ec_wt2_fp_loop_9", __LINE__, 1 + 1, "");
        for (i1 = 0; i1 < 2; i1++) {
          b_dv[i1] = static_cast<uint32_T>(cpu_yc->size[i1]);
        }
        profileLoopEnd();
        vstride = cpu_yc->size[0] * cpu_yc->size[1];
        cpu_yc->size[0] = static_cast<int32_T>(b_dv[0]);
        cpu_yc->size[1] = static_cast<int32_T>(b_dv[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_yc, vstride, &l_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(static_cast<int32_T>(b_dv[0]) *
                                static_cast<int32_T>(b_dv[1]) -
                            1),
            &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_yc, &gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt2_fp_kernel5#" MW_AT_LINE);
          ec_wt2_fp_kernel5<<<grid, block>>>(
              d_gpu_x,
              static_cast<int32_T>(b_dv[0]) * static_cast<int32_T>(b_dv[1]) - 1,
              gpu_yc);
        }
      }
      //  Downsample
      // 'ec_wt2_fp:66' if ds
      if (ds != 0.0) {
        int32_T cpu_lshift[2];
        int32_T cpu_sizeX[2];
        boolean_T sizeX_outdatedOnGpu;
        // 'ec_wt2_fp:67' yc = downsample(yc,ds);
        if (!(ds - 1.0 >= 0.0)) {
          d_y = nullptr;
          m2 = emlrtCreateCharArray(2, &iv1[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
          emlrtAssign(&d_y, m2);
          e_y = nullptr;
          m3 = emlrtCreateDoubleScalar(ds - 1.0);
          emlrtAssign(&e_y, m3);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(d_y, e_y, &c_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        dim = 1;
        if (cpu_yc->size[0] != 1) {
          dim = 0;
        }
        sizeX_outdatedOnGpu = false;
        lshift_outdatedOnCpu = false;
        lshift_outdatedOnGpu = false;
        profileLoopStart("ec_wt2_fp_loop_10", __LINE__, 1 + 1, "");
        for (i1 = 0; i1 < 2; i1++) {
          cpu_sizeX[i1] = cpu_yc->size[i1];
          sizeX_outdatedOnGpu = true;
          cpu_lshift[i1] = cpu_yc->size[i1];
          lshift_outdatedOnGpu = true;
        }
        profileLoopEnd();
        nvtxMarkA("#circshift#" MW_AT_LINE);
        coder::circshift(cpu_lshift, &lshift_outdatedOnCpu, *gpu_lshift,
                         &lshift_outdatedOnGpu, -dim);
        if (lshift_outdatedOnCpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if ((ds > 0.0) && (cpu_lshift[0] < 1)) {
          i1 = 1;
          nx = -1;
        } else {
          i1 = static_cast<int32_T>(ds);
          nx = cpu_lshift[0] - 1;
        }
        if (sizeX_outdatedOnGpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(*gpu_sizeX, cpu_sizeX, 8UL, hipMemcpyHostToDevice),
              __FILE__, __LINE__);
        }
        nvtxMarkA("#ec_wt2_fp_kernel6#" MW_AT_LINE);
        ec_wt2_fp_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(i1, nx, dim,
                                                                   *gpu_sizeX);
        dim = d_cpu_yc->size[0] * d_cpu_yc->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        d_cpu_yc->size[0] = div_s32(nx, i1) + 1;
        d_cpu_yc->size[1] = cpu_lshift[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(d_cpu_yc, dim, &k_emlrtRTEI);
        dim = nx / i1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, dim), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(d_cpu_yc, &b_gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt2_fp_kernel7#" MW_AT_LINE);
          ec_wt2_fp_kernel7<<<grid, block>>>(gpu_yc, cpu_lshift[0], i1, dim,
                                             cpu_lshift[1] - 1, b_gpu_yc,
                                             d_cpu_yc->size[0U]);
        }
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(
            hipMemcpy(cpu_sizeX, *gpu_sizeX, 8UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
        vstride = cpu_yc->size[0] * cpu_yc->size[1];
        cpu_yc->size[0] = cpu_sizeX[0];
        cpu_yc->size[1] = cpu_sizeX[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_yc, vstride, &n_emlrtRTEI);
        dim = cpu_sizeX[0] * cpu_sizeX[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(dim), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_yc, &gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt2_fp_kernel8#" MW_AT_LINE);
          ec_wt2_fp_kernel8<<<grid, block>>>(b_gpu_yc, dim, gpu_yc);
        }
      }
      vstride = y->data[k].f1->size[0] * y->data[k].f1->size[1];
      y->data[k].f1->size[0] = cpu_yc->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(y->data[k].f1, vstride, &j_emlrtRTEI);
      vstride = y->data[k].f1->size[0] * y->data[k].f1->size[1];
      y->data[k].f1->size[1] = cpu_yc->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(y->data[k].f1, vstride, &j_emlrtRTEI);
      profileLoopStart("ec_wt2_fp_loop_8", __LINE__,
                       (cpu_yc->size[0] * cpu_yc->size[1] - 1) + 1, "");
      for (vstride = 0; vstride < cpu_yc->size[0] * cpu_yc->size[1];
           vstride++) {
        if (yc_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(cpu_yc, &gpu_yc);
        }
        yc_outdatedOnCpu = false;
        y->data[k].f1->data[vstride].re = cpu_yc->data[vstride];
        y->data[k].f1->data[vstride].im = 0.0;
      }
      profileLoopEnd();
    } else {
      boolean_T validLaunchParams;
      boolean_T yc_outdatedOnCpu;
      // 'ec_wt2_fp:34' else
      // 'ec_wt2_fp:35' y{ch} = cwt_lfn(fb,x(:,ch),ds);
      //  Run CWT (complex)
      //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%% CWT
      // 'ec_wt2_fp:46' yc = wt(fb,xc)';
      vstride = d_cpu_x->size[0];
      d_cpu_x->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(d_cpu_x, vstride, &c_emlrtRTEI);
      dim = cpu_x->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(dim), &grid, &block, 2147483647U);
      if (x_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
      }
      x_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(d_cpu_x, &e_gpu_x, true);
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
      }
      x_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt2_fp_kernel9#" MW_AT_LINE);
        ec_wt2_fp_kernel9<<<grid, block>>>(k, gpu_x, dim, e_gpu_x,
                                           cpu_x->size[0U]);
      }
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(d_cpu_x, &e_gpu_x);
      nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
      coder::cwtfilterbank_wt(&fb, d_cpu_x, cpu_r);
      vstride = b_cpu_yc->size[0] * b_cpu_yc->size[1];
      b_cpu_yc->size[0] = cpu_r->size[1];
      b_cpu_yc->size[1] = cpu_r->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(b_cpu_yc, vstride, &e_emlrtRTEI);
      dim = cpu_r->size[0] - 1;
      nx = cpu_r->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(dim, nx), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_r, &gpu_r, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(b_cpu_yc, &c_gpu_yc, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r, cpu_r);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt2_fp_kernel10#" MW_AT_LINE);
        ec_wt2_fp_kernel10<<<grid, block>>>(
            gpu_r, nx, dim, c_gpu_yc, b_cpu_yc->size[0U], cpu_r->size[0U]);
      }
      yc_outdatedOnCpu = true;
      //  Downsample
      // 'ec_wt2_fp:49' if ds
      if (ds != 0.0) {
        int32_T cpu_lshift[2];
        int32_T cpu_sizeX[2];
        boolean_T sizeX_outdatedOnGpu;
        // 'ec_wt2_fp:50' yc = downsample(yc,ds);
        if (!(ds - 1.0 >= 0.0)) {
          b_y = nullptr;
          m = emlrtCreateCharArray(2, &iv[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
          emlrtAssign(&b_y, m);
          c_y = nullptr;
          m1 = emlrtCreateDoubleScalar(ds - 1.0);
          emlrtAssign(&c_y, m1);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(b_y, c_y, &c_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        dim = 1;
        if (b_cpu_yc->size[0] != 1) {
          dim = 0;
        }
        sizeX_outdatedOnGpu = false;
        lshift_outdatedOnCpu = false;
        lshift_outdatedOnGpu = false;
        profileLoopStart("ec_wt2_fp_loop_7", __LINE__, 1 + 1, "");
        for (i1 = 0; i1 < 2; i1++) {
          cpu_sizeX[i1] = b_cpu_yc->size[i1];
          sizeX_outdatedOnGpu = true;
          cpu_lshift[i1] = b_cpu_yc->size[i1];
          lshift_outdatedOnGpu = true;
        }
        profileLoopEnd();
        nvtxMarkA("#circshift#" MW_AT_LINE);
        coder::circshift(cpu_lshift, &lshift_outdatedOnCpu, *gpu_lshift,
                         &lshift_outdatedOnGpu, -dim);
        if (lshift_outdatedOnCpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        if ((ds > 0.0) && (cpu_lshift[0] < 1)) {
          i1 = 1;
          nx = -1;
        } else {
          i1 = static_cast<int32_T>(ds);
          nx = cpu_lshift[0] - 1;
        }
        if (sizeX_outdatedOnGpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(*gpu_sizeX, cpu_sizeX, 8UL, hipMemcpyHostToDevice),
              __FILE__, __LINE__);
        }
        nvtxMarkA("#ec_wt2_fp_kernel11#" MW_AT_LINE);
        ec_wt2_fp_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(i1, nx, dim,
                                                                    *gpu_sizeX);
        dim = c_cpu_yc->size[0] * c_cpu_yc->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        c_cpu_yc->size[0] = div_s32(nx, i1) + 1;
        c_cpu_yc->size[1] = cpu_lshift[1];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(c_cpu_yc, dim, &k_emlrtRTEI);
        dim = nx / i1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, dim), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(c_cpu_yc, &d_gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt2_fp_kernel12#" MW_AT_LINE);
          ec_wt2_fp_kernel12<<<grid, block>>>(c_gpu_yc, cpu_lshift[0], i1, dim,
                                              cpu_lshift[1] - 1, d_gpu_yc,
                                              c_cpu_yc->size[0U]);
        }
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(
            hipMemcpy(cpu_sizeX, *gpu_sizeX, 8UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
        vstride = b_cpu_yc->size[0] * b_cpu_yc->size[1];
        b_cpu_yc->size[0] = cpu_sizeX[0];
        b_cpu_yc->size[1] = cpu_sizeX[1];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(b_cpu_yc, vstride, &m_emlrtRTEI);
        dim = cpu_sizeX[0] * cpu_sizeX[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(dim), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(b_cpu_yc, &c_gpu_yc, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt2_fp_kernel13#" MW_AT_LINE);
          ec_wt2_fp_kernel13<<<grid, block>>>(d_gpu_yc, dim, c_gpu_yc);
        }
      }
      vstride = y->data[k].f1->size[0] * y->data[k].f1->size[1];
      y->data[k].f1->size[0] = b_cpu_yc->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(y->data[k].f1, vstride, &g_emlrtRTEI);
      vstride = y->data[k].f1->size[0] * y->data[k].f1->size[1];
      y->data[k].f1->size[1] = b_cpu_yc->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(y->data[k].f1, vstride, &g_emlrtRTEI);
      profileLoopStart("ec_wt2_fp_loop_6", __LINE__,
                       (b_cpu_yc->size[0] * b_cpu_yc->size[1] - 1) + 1, "");
      for (vstride = 0; vstride < b_cpu_yc->size[0] * b_cpu_yc->size[1];
           vstride++) {
        if (yc_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal_T(b_cpu_yc, &c_gpu_yc);
        }
        yc_outdatedOnCpu = false;
        y->data[k].f1->data[vstride] = b_cpu_yc->data[vstride];
      }
      profileLoopEnd();
    }
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_yc);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&c_cpu_yc);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&e_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r1);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_yc);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_yc);
  nvtxMarkA("#emxFreeStruct_cwtfilterbank#" MW_AT_LINE);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_yc);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&c_gpu_yc);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r1);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&e_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&d_gpu_yc);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_yc);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_sizeX), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_lshift), __FILE__, __LINE__);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_wt2_fp.cu)

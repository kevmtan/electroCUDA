//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt2_fp_terminate.cu
//
// Code generation for function 'ec_wt2_fp_terminate'
//

// Include files
#include "ec_wt2_fp_terminate.h"
#include "_coder_ec_wt2_fp_mex.h"
#include "ec_wt2_fp_data.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void emlrtExitTimeCleanupDtorFcn(const void *r);

// Function Definitions
static void emlrtExitTimeCleanupDtorFcn(const void *r)
{
  nvtxRangePushA("#fcn#emlrtExitTimeCleanupDtorFcn#" MW_AT_LOCATION);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  nvtxRangePop();
}

void ec_wt2_fp_atexit()
{
  nvtxRangePushA("#termFcn#ec_wt2_fp_atexit#" MW_AT_LOCATION);
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  try {
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&emlrtExitTimeCleanupDtorFcn,
                                      nullptr, nullptr, nullptr);
    emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
    emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
    emlrtExitTimeCleanup(&emlrtContextGlobal);
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  nvtxRangePop();
}

void ec_wt2_fp_terminate()
{
  hipError_t errCode;
  nvtxRangePushA("#fcn#ec_wt2_fp_terminate#" MW_AT_LOCATION);
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  nvtxRangePop();
}

// End of code generation (ec_wt2_fp_terminate.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt2_fp_initialize.cu
//
// Code generation for function 'ec_wt2_fp_initialize'
//

// Include files
#include "ec_wt2_fp_initialize.h"
#include "_coder_ec_wt2_fp_mex.h"
#include "ec_wt2_fp_data.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void ec_wt2_fp_once();

// Function Definitions
static void ec_wt2_fp_once()
{
  mex_InitInfAndNan();
  nvtxRangePushA("#initFcn#ec_wt2_fp_once#" MW_AT_LOCATION);
  // Initialize GPU by calling hipFree(nullptr)
  nvtxMarkA("#hipFree#" MW_AT_LINE);
  hipFree(nullptr);
  nvtxRangePop();
}

void ec_wt2_fp_initialize()
{
  nvtxRangePushA("#fcn#ec_wt2_fp_initialize#" MW_AT_LOCATION);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "wavelet_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "signal_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    nvtxMarkA("#ec_wt2_fp_once#" MW_AT_LINE);
    ec_wt2_fp_once();
  }
  nvtxRangePop();
}

// End of code generation (ec_wt2_fp_initialize.cu)

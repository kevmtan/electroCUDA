//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_detr_fp_terminate.cu
//
// Code generation for function 'ec_detr_fp_terminate'
//

// Include files
#include "ec_detr_fp_terminate.h"
#include "_coder_ec_detr_fp_mex.h"
#include "ec_detr_fp_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void checkCudaError(hipError_t errorCode);

static void emlrtExitTimeCleanupDtorFcn(const void *r);

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString);

// Function Definitions
static void checkCudaError(hipError_t errorCode)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errorCode != hipSuccess) {
    nvtxMarkA("#gpuThrowError#" MW_AT_LINE);
    gpuThrowError(errorCode, hipGetErrorName(errorCode),
                  hipGetErrorString(errorCode));
  }
  nvtxRangePop();
}

static void emlrtExitTimeCleanupDtorFcn(const void *r)
{
  nvtxRangePushA("#fcn#emlrtExitTimeCleanupDtorFcn#" MW_AT_LOCATION);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
  nvtxRangePop();
}

static void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                          const char_T *errorString)
{
  nvtxRangePushA("#fcn#gpuThrowError#" MW_AT_LOCATION);
  emlrtThinCUDAError(errorCode, (char_T *)errorName, (char_T *)errorString,
                     (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  nvtxRangePop();
}

void ec_detr_fp_atexit()
{
  nvtxRangePushA("#termFcn#ec_detr_fp_atexit#" MW_AT_LOCATION);
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  try {
    emlrtPushHeapReferenceStackR2021a(emlrtRootTLSGlobal, false, nullptr,
                                      (void *)&emlrtExitTimeCleanupDtorFcn,
                                      nullptr, nullptr, nullptr);
    emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
    emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
    emlrtExitTimeCleanup(&emlrtContextGlobal);
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
  cusolverDestroyWorkspace();
  cusolverEnsureDestruction();
  nvtxRangePop();
}

void ec_detr_fp_terminate()
{
  nvtxRangePushA("#fcn#ec_detr_fp_terminate#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  checkCudaError(hipGetLastError());
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  cublasEnsureDestruction();
  nvtxRangePop();
}

// End of code generation (ec_detr_fp_terminate.cu)

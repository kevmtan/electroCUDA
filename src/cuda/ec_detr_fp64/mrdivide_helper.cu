#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mrdivide_helper.cu
//
// Code generation for function 'mrdivide_helper'
//

// Include files
#include "mrdivide_helper.h"
#include "ec_detr_data.h"
#include "ec_detr_emxutil.h"
#include "ec_detr_mexutil.h"
#include "ec_detr_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "lapacke.h"
#include "hip/hip_math_constants.h"
#include <cmath>
#include <cstddef>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo md_emlrtRTEI{
    31,                // lineNo
    5,                 // colNo
    "mrdivide_helper", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "mrdivide_helper.m" // pName
};

static emlrtRTEInfo
    nd_emlrtRTEI{
        1,        // lineNo
        37,       // colNo
        "xgetrf", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgetrf.m" // pName
    };

static emlrtRTEInfo od_emlrtRTEI{
    44,                // lineNo
    32,                // colNo
    "mrdivide_helper", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "mrdivide_helper.m" // pName
};

static emlrtRTEInfo pd_emlrtRTEI{
    18,           // lineNo
    29,           // colNo
    "xgetrf_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrf_gpu.m" // pName
};

static emlrtRTEInfo qd_emlrtRTEI{
    44,                // lineNo
    35,                // colNo
    "mrdivide_helper", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "mrdivide_helper.m" // pName
};

static emlrtRTEInfo rd_emlrtRTEI{
    50,           // lineNo
    31,           // colNo
    "xgetrf_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrf_gpu.m" // pName
};

static emlrtRTEInfo
    sd_emlrtRTEI{
        61,       // lineNo
        9,        // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo td_emlrtRTEI{
    53,           // lineNo
    13,           // colNo
    "xgetrf_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrf_gpu.m" // pName
};

static emlrtRTEInfo
    ud_emlrtRTEI{
        92,       // lineNo
        22,       // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo
    vd_emlrtRTEI{
        105,      // lineNo
        1,        // colNo
        "xgeqp3", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeqp3.m" // pName
    };

static emlrtRTEInfo wd_emlrtRTEI{
    85,        // lineNo
    1,         // colNo
    "qrsolve", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/qrsolve.m" // pName
};

static emlrtRTEInfo xd_emlrtRTEI{
    119,       // lineNo
    5,         // colNo
    "qrsolve", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/qrsolve.m" // pName
};

static emlrtRTEInfo yd_emlrtRTEI{
    44,                // lineNo
    5,                 // colNo
    "mrdivide_helper", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "mrdivide_helper.m" // pName
};

static emlrtRTEInfo ae_emlrtRTEI{
    44,                // lineNo
    9,                 // colNo
    "mrdivide_helper", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "mrdivide_helper.m" // pName
};

static emlrtRTEInfo be_emlrtRTEI{
    25,                // lineNo
    14,                // colNo
    "mrdivide_helper", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "mrdivide_helper.m" // pName
};

static emlrtRTEInfo ce_emlrtRTEI{
    18,           // lineNo
    5,            // colNo
    "xgetrf_gpu", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+lapack/xgetrf_gpu.m" // pName
};

// Function Declarations
static void cusolverCheck(hipsolverStatus_t errCode, const char_T *file,
                          uint32_T b_line);

static void gpuEmxMemcpyGpuToCpu_int32_T(emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu);

static __global__ void mrdiv_kernel31(const int32_T b, const int32_T na,
                                      emxArray_real_T A, int32_T A_dim0);

static __global__ void mrdiv_kernel32(const emxArray_real_T B,
                                      const emxArray_int32_T jpvt,
                                      const int32_T na, emxArray_real_T Y);

// Function Definitions
static void cusolverCheck(hipsolverStatus_t errCode, const char_T *file,
                          uint32_T b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPSOLVER_STATUS_SUCCESS) {
    cusolverGetErrorName(errCode, &errName);
    cusolverGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

static void gpuEmxMemcpyGpuToCpu_int32_T(emxArray_int32_T *cpu,
                                         emxArray_int32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(int32_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

static __global__ __launch_bounds__(1024, 1) void mrdiv_kernel31(
    const int32_T b, const int32_T na, emxArray_real_T A, int32_T A_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(na) + 1UL) * (static_cast<uint64_T>(b) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T j;
    i = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    j = static_cast<int32_T>((idx - static_cast<uint64_T>(i)) /
                             (static_cast<uint64_T>(b) + 1UL));
    A.data[j * A_dim0 + i] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void mrdiv_kernel32(
    const emxArray_real_T B, const emxArray_int32_T jpvt, const int32_T na,
    emxArray_real_T Y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(na);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    Y.data[jpvt.data[i] - 1] = B.data[i];
  }
}

//
//
namespace coder {
namespace internal {
void mrdiv(emxArray_real_T *cpu_A, boolean_T *A_outdatedOnCpu,
           emxArray_real_T *gpu_A, boolean_T *A_outdatedOnGpu,
           const emxArray_real_T *B)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_ipiv_t;
  emxArray_int32_T gpu_jpvt;
  emxArray_int32_T *cpu_ipiv_t;
  emxArray_int32_T *cpu_jpvt;
  emxArray_int32_T *ipiv;
  emxArray_ptrdiff_t *jpvt_t;
  emxArray_real_T b_gpu_A;
  emxArray_real_T c_gpu_A;
  emxArray_real_T gpu_B;
  emxArray_real_T gpu_Y;
  emxArray_real_T gpu_tau;
  emxArray_real_T *b_cpu_A;
  emxArray_real_T *c_cpu_A;
  emxArray_real_T *cpu_B;
  emxArray_real_T *cpu_Y;
  emxArray_real_T *cpu_tau;
  int32_T cpu_minmn;
  int32_T *gpu_minmn;
  checkCudaError(mwCudaMalloc(&gpu_minmn, 4UL), __FILE__, __LINE__);
  gpuEmxReset_int32_T(&gpu_ipiv_t);
  gpuEmxReset_real_T(&gpu_tau);
  gpuEmxReset_int32_T(&gpu_jpvt);
  gpuEmxReset_real_T(&b_gpu_A);
  gpuEmxReset_real_T(&gpu_B);
  gpuEmxReset_real_T(&c_gpu_A);
  gpuEmxReset_real_T(&gpu_Y);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&cpu_Y, 1, &ae_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_A, 2, &be_emlrtRTEI, true);
  emxInit_real_T(&cpu_B, 1, &qd_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_A, 2, &be_emlrtRTEI, true);
  emxInit_int32_T(&ipiv, 2, &be_emlrtRTEI, true);
  emxInit_int32_T(&cpu_jpvt, 2, &be_emlrtRTEI, true);
  emxInit_real_T(&cpu_tau, 1, &be_emlrtRTEI, true);
  emxInit_int32_T(&cpu_ipiv_t, 1, &ce_emlrtRTEI, true);
  emxInit_ptrdiff_t(&jpvt_t, 1, &vd_emlrtRTEI, true);
  if ((cpu_A->size[1] == 0) || ((B->size[0] == 0) || (B->size[1] == 0))) {
    int32_T i;
    cpu_minmn = B->size[0];
    i = cpu_A->size[0] * cpu_A->size[1];
    cpu_A->size[0] = 1;
    cpu_A->size[1] = B->size[0];
    emxEnsureCapacity_real_T(cpu_A, i, &md_emlrtRTEI);
    if (cpu_minmn - 1 >= 0) {
      if (*A_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_A, gpu_A);
      }
      std::memset(&cpu_A->data[0], 0,
                  static_cast<uint32_T>(cpu_minmn) * sizeof(real_T));
      *A_outdatedOnCpu = false;
      *A_outdatedOnGpu = true;
    }
  } else if (B->size[0] == B->size[1]) {
    real_T tol;
    int32_T i;
    boolean_T B_needsGpuEnsureCapacity;
    boolean_T b_A_outdatedOnCpu;
    boolean_T b_A_outdatedOnGpu;
    b_A_outdatedOnGpu = false;
    i = c_cpu_A->size[0] * c_cpu_A->size[1];
    c_cpu_A->size[0] = B->size[0];
    c_cpu_A->size[1] = B->size[1];
    emxEnsureCapacity_real_T(c_cpu_A, i, &nd_emlrtRTEI);
    for (i = 0; i < B->size[0] * B->size[1]; i++) {
      c_cpu_A->data[i] = B->data[i];
      b_A_outdatedOnGpu = true;
    }
    i = cpu_ipiv_t->size[0];
    cpu_ipiv_t->size[0] = B->size[1];
    emxEnsureCapacity_int32_T(cpu_ipiv_t, i, &pd_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(c_cpu_A, &b_gpu_A, !b_A_outdatedOnGpu);
    if (b_A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_A, c_cpu_A);
    }
    cusolverCheck(
        hipsolverDnDgetrf_bufferSize(getCuSolverGlobalHandle(), B->size[1],
                                    B->size[1], (double *)&b_gpu_A.data[0],
                                    B->size[1], getCuSolverWorkspaceReq()),
        __FILE__, __LINE__);
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    b_A_outdatedOnCpu = false;
    gpuEmxEnsureCapacity_int32_T(cpu_ipiv_t, &gpu_ipiv_t, true);
    cusolverCheck(
        hipsolverDnDgetrf(getCuSolverGlobalHandle(), B->size[1], B->size[1],
                         (double *)&b_gpu_A.data[0], B->size[1],
                         static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                         &gpu_ipiv_t.data[0], gpu_minmn),
        __FILE__, __LINE__);
    B_needsGpuEnsureCapacity = true;
    b_A_outdatedOnGpu = false;
    i = ipiv->size[0] * ipiv->size[1];
    ipiv->size[0] = 1;
    ipiv->size[1] = cpu_ipiv_t->size[0];
    emxEnsureCapacity_int32_T(ipiv, i, &rd_emlrtRTEI);
    checkCudaError(
        hipMemcpy(&cpu_minmn, gpu_minmn, 4UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    if (cpu_minmn < 0) {
      int32_T maxmn;
      maxmn = c_cpu_A->size[0];
      cpu_minmn = c_cpu_A->size[1];
      i = c_cpu_A->size[0] * c_cpu_A->size[1];
      c_cpu_A->size[0] = maxmn;
      c_cpu_A->size[1] = cpu_minmn;
      emxEnsureCapacity_real_T(c_cpu_A, i, &td_emlrtRTEI);
      b_A_outdatedOnCpu = true;
      for (i = 0; i < maxmn * cpu_minmn; i++) {
        c_cpu_A->data[i] = rtNaN;
        b_A_outdatedOnGpu = true;
      }
      maxmn = cpu_ipiv_t->size[0] - 1;
      for (int32_T u0{0}; u0 <= maxmn; u0++) {
        ipiv->data[u0] = u0 + 1;
      }
    } else {
      int32_T maxmn;
      maxmn = cpu_ipiv_t->size[0] - 1;
      for (int32_T u0{0}; u0 <= maxmn; u0++) {
        if (B_needsGpuEnsureCapacity) {
          gpuEmxMemcpyGpuToCpu_int32_T(cpu_ipiv_t, &gpu_ipiv_t);
        }
        B_needsGpuEnsureCapacity = false;
        ipiv->data[u0] = cpu_ipiv_t->data[u0];
      }
    }
    tol = 1.0;
    if (b_A_outdatedOnCpu) {
      gpuEmxEnsureCapacity_real_T(c_cpu_A, &b_gpu_A, !b_A_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    if (b_A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_A, c_cpu_A);
    }
    if (*A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_A, cpu_A);
    }
    cublasCheck(hipblasDtrsm(getCublasGlobalHandle(), HIPBLAS_SIDE_RIGHT,
                            HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
                            HIPBLAS_DIAG_NON_UNIT, 1, B->size[1], (double *)&tol,
                            (double *)&b_gpu_A.data[0], B->size[1],
                            (double *)&gpu_A->data[0], 1),
                __FILE__, __LINE__);
    cublasCheck(hipblasDtrsm(getCublasGlobalHandle(), HIPBLAS_SIDE_RIGHT,
                            HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N,
                            HIPBLAS_DIAG_UNIT, 1, B->size[1], (double *)&tol,
                            (double *)&b_gpu_A.data[0], B->size[1],
                            (double *)&gpu_A->data[0], 1),
                __FILE__, __LINE__);
    *A_outdatedOnGpu = false;
    *A_outdatedOnCpu = true;
    cpu_minmn = B->size[1] - 2;
    for (int32_T u0{0}; u0 <= cpu_minmn; u0++) {
      i = ipiv->data[cpu_minmn - u0];
      if (i != (cpu_minmn - u0) + 1) {
        if (*A_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_A, gpu_A);
        }
        tol = cpu_A->data[cpu_minmn - u0];
        cpu_A->data[cpu_minmn - u0] = cpu_A->data[i - 1];
        cpu_A->data[i - 1] = tol;
        *A_outdatedOnCpu = false;
        *A_outdatedOnGpu = true;
      }
    }
  } else {
    ptrdiff_t info_t;
    real_T tol;
    int32_T i;
    int32_T maxmn;
    int32_T minmana;
    int32_T na;
    int32_T u0;
    boolean_T B_needsGpuEnsureCapacity;
    boolean_T B_outdatedOnGpu;
    boolean_T b_A_outdatedOnCpu;
    boolean_T b_A_outdatedOnGpu;
    boolean_T jpvt_outdatedOnGpu;
    boolean_T validLaunchParams;
    b_A_outdatedOnCpu = false;
    i = b_cpu_A->size[0] * b_cpu_A->size[1];
    b_cpu_A->size[0] = B->size[1];
    b_cpu_A->size[1] = B->size[0];
    emxEnsureCapacity_real_T(b_cpu_A, i, &od_emlrtRTEI);
    for (i = 0; i < B->size[0]; i++) {
      for (maxmn = 0; maxmn < B->size[1]; maxmn++) {
        b_cpu_A->data[maxmn + b_cpu_A->size[0] * i] =
            B->data[i + B->size[0] * maxmn];
      }
    }
    B_outdatedOnGpu = false;
    i = cpu_B->size[0];
    cpu_B->size[0] = cpu_A->size[1];
    emxEnsureCapacity_real_T(cpu_B, i, &qd_emlrtRTEI);
    for (i = 0; i < cpu_A->size[1]; i++) {
      if (*A_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_A, gpu_A);
      }
      *A_outdatedOnCpu = false;
      cpu_B->data[i] = cpu_A->data[i];
      B_outdatedOnGpu = true;
    }
    maxmn = b_cpu_A->size[0];
    na = b_cpu_A->size[1] - 1;
    jpvt_outdatedOnGpu = false;
    i = cpu_jpvt->size[0] * cpu_jpvt->size[1];
    cpu_jpvt->size[0] = 1;
    cpu_jpvt->size[1] = b_cpu_A->size[1];
    emxEnsureCapacity_int32_T(cpu_jpvt, i, &sd_emlrtRTEI);
    u0 = b_cpu_A->size[0];
    minmana = b_cpu_A->size[1];
    if (u0 <= minmana) {
      minmana = u0;
    }
    i = cpu_tau->size[0];
    cpu_tau->size[0] = minmana;
    emxEnsureCapacity_real_T(cpu_tau, i, &ud_emlrtRTEI);
    i = jpvt_t->size[0];
    jpvt_t->size[0] = b_cpu_A->size[1];
    emxEnsureCapacity_ptrdiff_t(jpvt_t, i, &vd_emlrtRTEI);
    for (i = 0; i < b_cpu_A->size[1]; i++) {
      cpu_jpvt->data[i] = 0;
      jpvt_outdatedOnGpu = true;
      jpvt_t->data[i] = (ptrdiff_t)0;
    }
    info_t = LAPACKE_dgeqp3(102, (ptrdiff_t)b_cpu_A->size[0],
                            (ptrdiff_t)b_cpu_A->size[1], &b_cpu_A->data[0],
                            (ptrdiff_t)b_cpu_A->size[0], &jpvt_t->data[0],
                            &cpu_tau->data[0]);
    b_A_outdatedOnGpu = true;
    if ((int32_T)info_t != 0) {
      int64_T b;
      mwGetLaunchParameters1D(computeNumIters(na, maxmn - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(b_cpu_A, &c_gpu_A, false);
      gpuEmxMemcpyCpuToGpu_real_T(&c_gpu_A, b_cpu_A);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        mrdiv_kernel31<<<grid, block>>>(maxmn - 1, na, c_gpu_A,
                                        b_cpu_A->size[0U]);
      }
      b_A_outdatedOnGpu = false;
      b_A_outdatedOnCpu = true;
      u0 = b_cpu_A->size[0];
      cpu_minmn = b_cpu_A->size[1];
      if (u0 <= cpu_minmn) {
        cpu_minmn = u0;
      }
      for (u0 = 0; u0 < cpu_minmn; u0++) {
        cpu_tau->data[u0] = rtNaN;
      }
      b = computeEndIdx(static_cast<int64_T>(cpu_minmn + 1),
                        static_cast<int64_T>(minmana), 1L);
      for (int64_T k{0L}; k <= b; k++) {
        cpu_tau->data[static_cast<int32_T>((cpu_minmn + 1) + k) - 1] = 0.0;
      }
      for (u0 = 0; u0 <= na; u0++) {
        cpu_jpvt->data[u0] = u0 + 1;
        jpvt_outdatedOnGpu = true;
      }
    } else {
      for (u0 = 0; u0 <= na; u0++) {
        cpu_jpvt->data[u0] = (int32_T)jpvt_t->data[u0];
        jpvt_outdatedOnGpu = true;
      }
    }
    na = -1;
    if (b_cpu_A->size[0] < b_cpu_A->size[1]) {
      cpu_minmn = b_cpu_A->size[0];
      maxmn = b_cpu_A->size[1];
    } else {
      cpu_minmn = b_cpu_A->size[1];
      maxmn = b_cpu_A->size[0];
    }
    if (b_A_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real_T(b_cpu_A, &c_gpu_A);
    }
    tol = std::fmin(1.4901161193847656E-8,
                    2.2204460492503131E-15 * static_cast<real_T>(maxmn)) *
          std::abs(b_cpu_A->data[0]);
    while ((na + 1 < cpu_minmn) &&
           (!(std::abs(b_cpu_A->data[(na + b_cpu_A->size[0] * (na + 1)) + 1]) <=
              tol))) {
      na++;
    }
    b_A_outdatedOnCpu = false;
    i = cpu_Y->size[0];
    cpu_Y->size[0] = b_cpu_A->size[1];
    emxEnsureCapacity_real_T(cpu_Y, i, &wd_emlrtRTEI);
    for (i = 0; i < b_cpu_A->size[1]; i++) {
      cpu_Y->data[i] = 0.0;
      b_A_outdatedOnCpu = true;
    }
    cpu_minmn = 0;
    u0 = b_cpu_A->size[0];
    maxmn = b_cpu_A->size[1];
    if (u0 <= maxmn) {
      maxmn = u0;
    }
    gpuEmxEnsureCapacity_real_T(b_cpu_A, &c_gpu_A, !b_A_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_tau, &gpu_tau, false);
    gpuEmxEnsureCapacity_real_T(cpu_B, &gpu_B, !B_outdatedOnGpu);
    if (b_A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&c_gpu_A, b_cpu_A);
    }
    gpuEmxMemcpyCpuToGpu_real_T(&gpu_tau, cpu_tau);
    if (B_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_B, cpu_B);
    }
    cusolverCheck(hipsolverDnDormqr_bufferSize(
                      getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                      cpu_B->size[0], 1, maxmn, (double *)&c_gpu_A.data[0],
                      b_cpu_A->size[0], (double *)&gpu_tau.data[0],
                      (double *)&gpu_B.data[0], cpu_B->size[0],
                      getCuSolverWorkspaceReq()),
                  __FILE__, __LINE__);
    setCuSolverWorkspaceTypeSize(8);
    cusolverInitWorkspace();
    B_needsGpuEnsureCapacity = false;
    checkCudaError(
        hipMemcpy(gpu_minmn, &cpu_minmn, 4UL, hipMemcpyHostToDevice),
        __FILE__, __LINE__);
    cusolverCheck(hipsolverDnDormqr(
                      getCuSolverGlobalHandle(), HIPBLAS_SIDE_LEFT, HIPBLAS_OP_T,
                      cpu_B->size[0], 1, maxmn, (double *)&c_gpu_A.data[0],
                      b_cpu_A->size[0], (double *)&gpu_tau.data[0],
                      (double *)&gpu_B.data[0], cpu_B->size[0],
                      static_cast<real_T *>(getCuSolverWorkspaceBuff()),
                      *getCuSolverWorkspaceReq(), gpu_minmn),
                  __FILE__, __LINE__);
    B_outdatedOnGpu = false;
    checkCudaError(
        hipMemcpy(&cpu_minmn, gpu_minmn, 4UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
    if (cpu_minmn < 0) {
      maxmn = cpu_B->size[0];
      i = cpu_B->size[0];
      cpu_B->size[0] = maxmn;
      emxEnsureCapacity_real_T(cpu_B, i, &xd_emlrtRTEI);
      B_needsGpuEnsureCapacity = true;
      for (i = 0; i < maxmn; i++) {
        cpu_B->data[i] = rtNaN;
        B_outdatedOnGpu = true;
      }
    }
    mwGetLaunchParameters1D(computeNumIters(na), &grid, &block, 2147483647U);
    if (B_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(cpu_B, &gpu_B, !B_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_int32_T(cpu_jpvt, &gpu_jpvt, !jpvt_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_Y, &gpu_Y, !b_A_outdatedOnCpu);
    if (B_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_B, cpu_B);
    }
    if (jpvt_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_jpvt, cpu_jpvt);
    }
    if (b_A_outdatedOnCpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_Y, cpu_Y);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      mrdiv_kernel32<<<grid, block>>>(gpu_B, gpu_jpvt, na, gpu_Y);
    }
    b_A_outdatedOnCpu = true;
    for (u0 = 0; u0 <= na; u0++) {
      i = cpu_jpvt->data[na - u0];
      if (b_A_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_Y, &gpu_Y);
      }
      cpu_Y->data[i - 1] /=
          b_cpu_A->data[(na - u0) + b_cpu_A->size[0] * (na - u0)];
      b_A_outdatedOnCpu = false;
      maxmn = na - u0;
      for (cpu_minmn = 0; cpu_minmn < maxmn; cpu_minmn++) {
        cpu_Y->data[cpu_jpvt->data[cpu_minmn] - 1] -=
            cpu_Y->data[cpu_jpvt->data[na - u0] - 1] *
            b_cpu_A->data[cpu_minmn + b_cpu_A->size[0] * (na - u0)];
      }
    }
    i = cpu_A->size[0] * cpu_A->size[1];
    cpu_A->size[0] = 1;
    cpu_A->size[1] = cpu_Y->size[0];
    emxEnsureCapacity_real_T(cpu_A, i, &yd_emlrtRTEI);
    for (i = 0; i < cpu_Y->size[0]; i++) {
      if (*A_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_A, gpu_A);
      }
      if (b_A_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_Y, &gpu_Y);
      }
      b_A_outdatedOnCpu = false;
      cpu_A->data[i] = cpu_Y->data[i];
      *A_outdatedOnCpu = false;
      *A_outdatedOnGpu = true;
    }
  }
  emxFree_ptrdiff_t(&jpvt_t);
  emxFree_int32_T(&cpu_ipiv_t);
  emxFree_real_T(&cpu_tau);
  emxFree_int32_T(&cpu_jpvt);
  emxFree_int32_T(&ipiv);
  emxFree_real_T(&c_cpu_A);
  emxFree_real_T(&cpu_B);
  emxFree_real_T(&b_cpu_A);
  emxFree_real_T(&cpu_Y);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_Y);
  gpuEmxFree_real_T(&c_gpu_A);
  gpuEmxFree_real_T(&gpu_B);
  gpuEmxFree_real_T(&b_gpu_A);
  gpuEmxFree_int32_T(&gpu_jpvt);
  gpuEmxFree_real_T(&gpu_tau);
  gpuEmxFree_int32_T(&gpu_ipiv_t);
  checkCudaError(mwCudaFree(gpu_minmn), __FILE__, __LINE__);
}

} // namespace internal
} // namespace coder

// End of code generation (mrdivide_helper.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_detr.cu
//
// Code generation for function 'ec_detr'
//

// Include files
#include "ec_detr.h"
#include "ec_detr_data.h"
#include "ec_detr_emxutil.h"
#include "ec_detr_mexutil.h"
#include "ec_detr_types.h"
#include "eig.h"
#include "mrdivide_helper.h"
#include "mtimes.h"
#include "rt_nonfinite.h"
#include "sum.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hip/hip_math_constants.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo ub_emlrtRTEI{
    32,                                                  // lineNo
    24,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo vb_emlrtRTEI{
    58,                                                  // lineNo
    23,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo wb_emlrtRTEI{
    49,                                                                // lineNo
    20,                                                                // colNo
    "linspace",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/linspace.m" // pName
};

static emlrtRTEInfo xb_emlrtRTEI{
    41,                                                  // lineNo
    41,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo yb_emlrtRTEI{
    41,                                                  // lineNo
    49,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo ac_emlrtRTEI{
    80,                                                  // lineNo
    20,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo bc_emlrtRTEI{
    42,                                                  // lineNo
    28,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo cc_emlrtRTEI{
    124,                                                 // lineNo
    10,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo dc_emlrtRTEI{
    109,                                                 // lineNo
    1,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo ec_emlrtRTEI{
    110,                                                 // lineNo
    1,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo
    fc_emlrtRTEI{
        44,       // lineNo
        36,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo gc_emlrtRTEI{
    112,                                                 // lineNo
    1,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo hc_emlrtRTEI{
    109,                                                           // lineNo
    24,                                                            // colNo
    "diag",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/diag.m" // pName
};

static emlrtRTEInfo ic_emlrtRTEI{
    100,                                                           // lineNo
    5,                                                             // colNo
    "diag",                                                        // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/elmat/diag.m" // pName
};

static emlrtRTEInfo jc_emlrtRTEI{
    1,                                                   // lineNo
    24,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo kc_emlrtRTEI{
    114,                                                 // lineNo
    5,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo lc_emlrtRTEI{
    114,                                                 // lineNo
    1,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo mc_emlrtRTEI{
    115,                                                 // lineNo
    6,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo nc_emlrtRTEI{
    108,                                                 // lineNo
    5,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo
    oc_emlrtRTEI{
        94,                  // lineNo
        5,                   // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo
    pc_emlrtRTEI{
        44,       // lineNo
        48,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo qc_emlrtRTEI{
    83,                                                  // lineNo
    5,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    86,                                                  // lineNo
    5,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo
    sc_emlrtRTEI{
        126,       // lineNo
        34,        // colNo
        "vvarstd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
        "vvarstd.m" // pName
    };

static emlrtRTEInfo tc_emlrtRTEI{
    30,                    // lineNo
    21,                    // colNo
    "applyScalarFunction", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/"
    "applyScalarFunction.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    1,                                                   // lineNo
    15,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    37,                                                  // lineNo
    5,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo wc_emlrtRTEI{
    59,                                                  // lineNo
    1,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo xc_emlrtRTEI{
    80,                                                  // lineNo
    1,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo yc_emlrtRTEI{
    83,                                                  // lineNo
    22,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo ad_emlrtRTEI{
    111,                                                 // lineNo
    13,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo bd_emlrtRTEI{
    111,                                                 // lineNo
    2,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo cd_emlrtRTEI{
    113,                                                 // lineNo
    10,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo dd_emlrtRTEI{
    114,                                                 // lineNo
    9,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo ed_emlrtRTEI{
    116,                                                 // lineNo
    6,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    116,                                                 // lineNo
    16,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo gd_emlrtRTEI{
    117,                                                 // lineNo
    8,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo hd_emlrtRTEI{
    117,                                                 // lineNo
    5,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo id_emlrtRTEI{
    88,                                                  // lineNo
    7,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo
    jd_emlrtRTEI{
        126,       // lineNo
        9,         // colNo
        "vvarstd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
        "vvarstd.m" // pName
    };

static emlrtRTEInfo kd_emlrtRTEI{
    124,                                                 // lineNo
    6,                                                   // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

static emlrtRTEInfo
    ld_emlrtRTEI{
        94,                  // lineNo
        9,                   // colNo
        "eml_mtimes_helper", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/"
        "eml_mtimes_helper.m" // pName
    };

static emlrtRTEInfo de_emlrtRTEI{
    41,                                                  // lineNo
    10,                                                  // colNo
    "ec_detr",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_detr.m" // pName
};

// Function Declarations
static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             const emxArray_real_T *in3,
                             const emxArray_real_T *in4);

static void binary_expand_op_1(emxArray_real_T *in1, const emxArray_real_T *in2,
                               const emxArray_real_T *in3, real_T in4);

static void binary_expand_op_2(emxArray_real_T *in1, const emxArray_real_T *in2,
                               real_T in3, const emxArray_real_T *in4);

static void binary_expand_op_3(emxArray_real_T *in1, const emxArray_real_T *in2,
                               const emxArray_real_T *in3);

static void binary_expand_op_4(emxArray_real_T *in1, const emxArray_real_T *in2,
                               const emxArray_real_T *in3);

static void binary_expand_op_5(emxArray_real_T *in1, int32_T in2,
                               const emxArray_real_T *in3);

static __global__ void ec_detr_kernel1(const int32_T nFrames,
                                       emxArray_real_T lin);

static __global__ void ec_detr_kernel10(const emxArray_real_T dv1,
                                        const emxArray_real_T regs,
                                        const int32_T b_regs,
                                        const int32_T c_regs,
                                        emxArray_real_T d_regs,
                                        int32_T regs_dim0, int32_T b_regs_dim0);

static __global__ void ec_detr_kernel11(const real_T y, const emxArray_real_T r,
                                        const emxArray_real_T regs,
                                        const int32_T b_regs,
                                        const int32_T c_regs,
                                        emxArray_real_T b_r, int32_T r_dim0,
                                        int32_T regs_dim0);

static __global__ void ec_detr_kernel12(const emxArray_real_T dv1,
                                        const emxArray_real_T r,
                                        const int32_T b_r, const int32_T c_r,
                                        emxArray_real_T r1, int32_T r1_dim0,
                                        int32_T r_dim0);

static __global__ void ec_detr_kernel13(const emxArray_real_T r1,
                                        const int32_T b_r1, const int32_T c_r1,
                                        emxArray_real_T r2, int32_T r2_dim0,
                                        int32_T r1_dim0);

static __global__ void ec_detr_kernel14(const emxArray_creal_T V,
                                        const int32_T b_V, emxArray_real_T c_V);

static __global__ void ec_detr_kernel15(const emxArray_int32_T iv,
                                        const emxArray_real_T V,
                                        const int32_T b, const int32_T b_iv,
                                        emxArray_real_T b_V, int32_T V_dim0,
                                        int32_T b_V_dim0);

static __global__ void ec_detr_kernel16(const emxArray_real_T V,
                                        const int32_T b_V, emxArray_real_T c_V);

static __global__ void ec_detr_kernel17(const emxArray_real_T dv1,
                                        const real_T mn,
                                        const emxArray_real_T dv,
                                        const int32_T b_dv, emxArray_real_T A);

static __global__ void ec_detr_kernel18(const int32_T b, emxArray_real_T y);

static __global__ void ec_detr_kernel19(const emxArray_real_T r1,
                                        const int32_T b_r1, const int32_T c_r1,
                                        emxArray_real_T r3, int32_T r3_dim0,
                                        int32_T r1_dim0);

static __global__ void ec_detr_kernel2(emxArray_real_T lin);

static __global__ void ec_detr_kernel20(const emxArray_real_T y,
                                        const int32_T b_y, emxArray_real_T c_y);

static __global__ void ec_detr_kernel21(const real_T mn,
                                        const emxArray_real_T dv,
                                        const emxArray_real_T dv2,
                                        const int32_T b_dv2, emxArray_real_T z);

static __global__ void ec_detr_kernel22(const emxArray_real_T dv1,
                                        const emxArray_real_T z,
                                        const emxArray_real_T dv,
                                        const int32_T b_dv, emxArray_real_T d);

static __global__ void ec_detr_kernel23(const emxArray_real_T d,
                                        const int32_T b, emxArray_real_T y);

static __global__ void ec_detr_kernel24(const real_T d2scaled,
                                        const emxArray_real_T y,
                                        const int32_T k, emxArray_real_T dv1);

static __global__ void ec_detr_kernel25(const emxArray_real_T d, real_T *y);

static __global__ void ec_detr_kernel26(const real_T d2scaled,
                                        const emxArray_real_T d,
                                        const int32_T nx,
                                        emxArray_real_T absdiff);

static __global__ void ec_detr_kernel27(const emxArray_creal_T D,
                                        const int32_T nz, emxArray_creal_T d,
                                        int32_T D_dim0);

static __global__ void ec_detr_kernel28(const emxArray_real_T z,
                                        const int32_T ch, const int32_T b,
                                        emxArray_real_T x, int32_T x_dim0);

static __global__ void ec_detr_kernel29(const emxArray_real_T dv1,
                                        const int32_T ch, const int32_T b,
                                        emxArray_real_T w, int32_T w_dim0);

static __global__ void ec_detr_kernel3(const real_T d2scaled, const int32_T b,
                                       emxArray_real_T lin, int32_T lin_dim1);

static __global__ void ec_detr_kernel30(const int32_T ch,
                                        const emxArray_real_T w,
                                        const int32_T b_w, emxArray_boolean_T x,
                                        int32_T w_dim0);

static __global__ void ec_detr_kernel4(emxArray_real_T lin, int32_T lin_dim1);

static __global__ void ec_detr_kernel5(const emxArray_real_T lin,
                                       const int32_T k, const int32_T b,
                                       emxArray_real_T regs, int32_T regs_dim0);

static __global__ void ec_detr_kernel6(const int32_T ch,
                                       const emxArray_real_T x,
                                       const int32_T b_x, emxArray_real_T dv,
                                       int32_T x_dim0);

static __global__ void ec_detr_kernel7(const int32_T ch,
                                       const emxArray_real_T w,
                                       const int32_T b_w, emxArray_real_T dv1,
                                       int32_T w_dim0);

static __global__ void ec_detr_kernel8(const emxArray_real_T dv1,
                                       const emxArray_real_T dv,
                                       const int32_T b_dv, emxArray_real_T x);

static __global__ void ec_detr_kernel9(const emxArray_real_T dv1,
                                       const emxArray_real_T dv,
                                       const int32_T b_dv, emxArray_real_T x);

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy);

static void gpuEmxEnsureCapacity_creal_T(const emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu,
                                         boolean_T needsCopy);

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxFree_creal_T(emxArray_creal_T *gpu);

static void gpuEmxMemcpyCpuToGpu_creal_T(emxArray_creal_T *gpu,
                                         const emxArray_creal_T *cpu);

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu);

static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu);

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu);

static void gpuEmxReset_creal_T(emxArray_creal_T *gpu);

static void times(emxArray_real_T *in1, const emxArray_real_T *in2,
                  const emxArray_real_T *in3);

// Function Definitions
static void binary_expand_op(emxArray_real_T *in1, const emxArray_real_T *in2,
                             const emxArray_real_T *in3,
                             const emxArray_real_T *in4)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  int32_T stride_2_0;
  i = in1->size[0];
  if (in4->size[0] == 1) {
    if (in3->size[0] == 1) {
      in1->size[0] = in2->size[0];
    } else {
      in1->size[0] = in3->size[0];
    }
  } else {
    in1->size[0] = in4->size[0];
  }
  emxEnsureCapacity_real_T(in1, i, &rc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  stride_2_0 = (in4->size[0] != 1);
  if (in4->size[0] == 1) {
    if (in3->size[0] == 1) {
      b = in2->size[0];
    } else {
      b = in3->size[0];
    }
  } else {
    b = in4->size[0];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] = (in2->data[i * stride_0_0] - in3->data[i * stride_1_0]) *
                   in4->data[i * stride_2_0];
  }
}

static void binary_expand_op_1(emxArray_real_T *in1, const emxArray_real_T *in2,
                               const emxArray_real_T *in3, real_T in4)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  i = in1->size[0];
  if (in3->size[0] == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3->size[0];
  }
  emxEnsureCapacity_real_T(in1, i, &qc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  if (in3->size[0] == 1) {
    b = in2->size[0];
  } else {
    b = in3->size[0];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] =
        in2->data[i * stride_0_0] +
        (in3->data[i * stride_1_0] - (in3->data[i * stride_1_0] - in4));
  }
}

static void binary_expand_op_2(emxArray_real_T *in1, const emxArray_real_T *in2,
                               real_T in3, const emxArray_real_T *in4)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  if (in4->size[0] == 1) {
    in1->size[1] = in2->size[0];
  } else {
    in1->size[1] = in4->size[0];
  }
  emxEnsureCapacity_real_T(in1, i, &nc_emlrtRTEI);
  stride_0_1 = (in2->size[0] != 1);
  stride_1_1 = (in4->size[0] != 1);
  if (in4->size[0] == 1) {
    b = in2->size[0];
  } else {
    b = in4->size[0];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] =
        (in2->data[i * stride_0_1] - in3) * in4->data[i * stride_1_1];
  }
}

static void binary_expand_op_3(emxArray_real_T *in1, const emxArray_real_T *in2,
                               const emxArray_real_T *in3)
{
  int32_T i;
  int32_T in3_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  in3_idx_0 = in3->size[0];
  i = in1->size[0] * in1->size[1];
  if (in3_idx_0 == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3_idx_0;
  }
  in1->size[1] = in2->size[1];
  emxEnsureCapacity_real_T(in1, i, &ec_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3_idx_0 != 1);
  for (i = 0; i < in2->size[1]; i++) {
    int32_T b;
    if (in3_idx_0 == 1) {
      b = in2->size[0];
    } else {
      b = in3_idx_0;
    }
    for (int32_T i1{0}; i1 < b; i1++) {
      in1->data[i1 + in1->size[0] * i] =
          in2->data[i1 * stride_0_0 + in2->size[0] * i] *
          in3->data[i1 * stride_1_0];
    }
  }
}

static void binary_expand_op_4(emxArray_real_T *in1, const emxArray_real_T *in2,
                               const emxArray_real_T *in3)
{
  emxArray_real_T *b_in2;
  int32_T i;
  int32_T in3_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  in3_idx_0 = in3->size[0];
  emxInit_real_T(&b_in2, 2, &cc_emlrtRTEI, true);
  i = b_in2->size[0] * b_in2->size[1];
  if (in3_idx_0 == 1) {
    b_in2->size[0] = in2->size[0];
  } else {
    b_in2->size[0] = in3_idx_0;
  }
  b_in2->size[1] = in2->size[1];
  emxEnsureCapacity_real_T(b_in2, i, &cc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3_idx_0 != 1);
  for (i = 0; i < in2->size[1]; i++) {
    int32_T b;
    if (in3_idx_0 == 1) {
      b = in2->size[0];
    } else {
      b = in3_idx_0;
    }
    for (int32_T i1{0}; i1 < b; i1++) {
      b_in2->data[i1 + b_in2->size[0] * i] =
          in2->data[i1 * stride_0_0 + in2->size[0] * i] *
          in3->data[i1 * stride_1_0];
    }
  }
  coder::sum(b_in2, in1);
  emxFree_real_T(&b_in2);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

static void binary_expand_op_5(emxArray_real_T *in1, int32_T in2,
                               const emxArray_real_T *in3)
{
  emxArray_real_T *b_in1;
  int32_T i;
  int32_T in1_idx_0;
  int32_T stride_0_0;
  int32_T stride_1_0;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  in1_idx_0 = in1->size[0];
  emxInit_real_T(&b_in1, 1, &de_emlrtRTEI, true);
  i = b_in1->size[0];
  b_in1->size[0] = in1_idx_0;
  emxEnsureCapacity_real_T(b_in1, i, &de_emlrtRTEI);
  stride_0_0 = (in1->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  for (i = 0; i < in1_idx_0; i++) {
    b_in1->data[i] = in1->data[i * stride_0_0 + in1->size[0] * in2] -
                     in3->data[i * stride_1_0];
  }
  for (i = 0; i < b_in1->size[0]; i++) {
    in1->data[i + in1->size[0] * in2] = b_in1->data[i];
  }
  emxFree_real_T(&b_in1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

static __global__
    __launch_bounds__(32, 1) void ec_detr_kernel1(const int32_T nFrames,
                                                  emxArray_real_T lin)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    lin.data[nFrames - 1] = 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel10(
    const emxArray_real_T dv1, const emxArray_real_T regs, const int32_T b_regs,
    const int32_T c_regs, emxArray_real_T d_regs, int32_T regs_dim0,
    int32_T b_regs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_regs) + 1UL) *
                (static_cast<uint64_T>(b_regs) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_regs) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_regs) + 1UL));
    d_regs.data[k + regs_dim0 * i] =
        regs.data[k + b_regs_dim0 * i] * dv1.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel11(
    const real_T y, const emxArray_real_T r, const emxArray_real_T regs,
    const int32_T b_regs, const int32_T c_regs, emxArray_real_T b_r,
    int32_T r_dim0, int32_T regs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_regs) + 1UL) *
                (static_cast<uint64_T>(b_regs) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_regs) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_regs) + 1UL));
    b_r.data[k + r_dim0 * i] = regs.data[k + regs_dim0 * i] - r.data[i] / y;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel12(
    const emxArray_real_T dv1, const emxArray_real_T r, const int32_T b_r,
    const int32_T c_r, emxArray_real_T r1, int32_T r1_dim0, int32_T r_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_r) + 1UL) * (static_cast<uint64_T>(b_r) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r) + 1UL));
    r1.data[k + r1_dim0 * i] = r.data[k + r_dim0 * i] * dv1.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel13(
    const emxArray_real_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_real_T r2, int32_T r2_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r1) + 1UL));
    r2.data[k + r2_dim0 * i] = r1.data[i + r1_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel14(
    const emxArray_creal_T V, const int32_T b_V, emxArray_real_T c_V)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_V);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_V.data[i] = V.data[i].re;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel15(
    const emxArray_int32_T iv, const emxArray_real_T V, const int32_T b,
    const int32_T b_iv, emxArray_real_T b_V, int32_T V_dim0, int32_T b_V_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b_iv) + 1UL) * (static_cast<uint64_T>(b) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b) + 1UL));
    b_V.data[k + V_dim0 * i] = V.data[k + b_V_dim0 * iv.data[i]];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel16(
    const emxArray_real_T V, const int32_T b_V, emxArray_real_T c_V)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_V);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_V.data[i] = V.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel17(
    const emxArray_real_T dv1, const real_T mn, const emxArray_real_T dv,
    const int32_T b_dv, emxArray_real_T A)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    A.data[i] = (dv.data[i] - mn) * dv1.data[i];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void ec_detr_kernel18(const int32_T b,
                                                             emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    y.data[i] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel19(
    const emxArray_real_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_real_T r3, int32_T r3_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    i = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                             (static_cast<uint64_T>(b_r1) + 1UL));
    r3.data[k + r3_dim0 * i] = r1.data[i + r1_dim0 * k];
  }
}

static __global__ __launch_bounds__(32,
                                    1) void ec_detr_kernel2(emxArray_real_T lin)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    lin.data[0] = -1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel20(
    const emxArray_real_T y, const int32_T b_y, emxArray_real_T c_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    c_y.data[i] = y.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel21(
    const real_T mn, const emxArray_real_T dv, const emxArray_real_T dv2,
    const int32_T b_dv2, emxArray_real_T z)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_dv2);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    z.data[i] = dv2.data[i] + (dv.data[i] - (dv.data[i] - mn));
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel22(
    const emxArray_real_T dv1, const emxArray_real_T z,
    const emxArray_real_T dv, const int32_T b_dv, emxArray_real_T d)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    d.data[i] = (dv.data[i] - z.data[i]) * dv1.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel23(
    const emxArray_real_T d, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = fabs(d.data[k]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel24(
    const real_T d2scaled, const emxArray_real_T y, const int32_T k,
    emxArray_real_T dv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(k);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    b_k = static_cast<int32_T>(idx);
    if (y.data[b_k] > d2scaled) {
      dv1.data[b_k] = 0.0;
    }
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_detr_kernel25(const emxArray_real_T d,
                                                   real_T *y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    if ((static_cast<boolean_T>(!static_cast<int32_T>(isinf(d.data[0])))) &&
        (static_cast<boolean_T>(!static_cast<int32_T>(isnan(d.data[0]))))) {
      *y = 0.0;
    } else {
      *y = HIP_NAN;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel26(
    const real_T d2scaled, const emxArray_real_T d, const int32_T nx,
    emxArray_real_T absdiff)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nx);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    absdiff.data[k] = fabs(d.data[k] - d2scaled);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel27(
    const emxArray_creal_T D, const int32_T nz, emxArray_creal_T d,
    int32_T D_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nz);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    d.data[k] = D.data[k + D_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel28(
    const emxArray_real_T z, const int32_T ch, const int32_T b,
    emxArray_real_T x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i + x_dim0 * ch] -= z.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel29(
    const emxArray_real_T dv1, const int32_T ch, const int32_T b,
    emxArray_real_T w, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    w.data[i + w_dim0 * ch] = dv1.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel3(
    const real_T d2scaled, const int32_T b, emxArray_real_T lin,
    int32_T lin_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    lin.data[k + 1] =
        static_cast<real_T>((((k + 2) << 1) - lin_dim1) - 1) * d2scaled;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel30(
    const int32_T ch, const emxArray_real_T w, const int32_T b_w,
    emxArray_boolean_T x, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_w);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = static_cast<boolean_T>(
        !static_cast<int32_T>(w.data[i + w_dim0 * ch] != 0.0));
  }
}

static __global__ __launch_bounds__(32,
                                    1) void ec_detr_kernel4(emxArray_real_T lin,
                                                            int32_T lin_dim1)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    lin.data[lin_dim1 >> 1] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel5(
    const emxArray_real_T lin, const int32_T k, const int32_T b,
    emxArray_real_T regs, int32_T regs_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    b_k = static_cast<int32_T>(idx);
    regs.data[b_k + regs_dim0 * k] =
        pow(lin.data[b_k], static_cast<real_T>(k) + 1.0);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel6(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T dv, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    dv.data[i] = x.data[i + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel7(
    const int32_T ch, const emxArray_real_T w, const int32_T b_w,
    emxArray_real_T dv1, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_w);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    dv1.data[i] = w.data[i + w_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel8(
    const emxArray_real_T dv1, const emxArray_real_T dv, const int32_T b_dv,
    emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = dv.data[i] * dv1.data[i];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_detr_kernel9(
    const emxArray_real_T dv1, const emxArray_real_T dv, const int32_T b_dv,
    emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i;
    i = static_cast<int32_T>(idx);
    x.data[i] = dv.data[i] * dv1.data[i];
  }
}

static void gpuEmxEnsureCapacity_boolean_T(const emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu,
                                           boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  boolean_T *newData;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(boolean_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(boolean_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxEnsureCapacity_creal_T(const emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu,
                                         boolean_T needsCopy)
{
  creal_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(creal_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(creal_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
}

static void gpuEmxFree_boolean_T(emxArray_boolean_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxFree_creal_T(emxArray_creal_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_creal_T(emxArray_creal_T *gpu,
                                         const emxArray_creal_T *cpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(creal_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_boolean_T(emxArray_boolean_T *cpu,
                                           emxArray_boolean_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(
      hipMemcpy(cpu->data, gpu->data,
                 static_cast<uint32_T>(actualSize) * sizeof(boolean_T),
                 hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
}

static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(creal_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
}

static void gpuEmxReset_boolean_T(emxArray_boolean_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_boolean_T));
}

static void gpuEmxReset_creal_T(emxArray_creal_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_creal_T));
}

static void times(emxArray_real_T *in1, const emxArray_real_T *in2,
                  const emxArray_real_T *in3)
{
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_1_0;
  i = in1->size[0];
  if (in3->size[0] == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3->size[0];
  }
  emxEnsureCapacity_real_T(in1, i, &cc_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_1_0 = (in3->size[0] != 1);
  if (in3->size[0] == 1) {
    b = in2->size[0];
  } else {
    b = in3->size[0];
  }
  for (i = 0; i < b; i++) {
    in1->data[i] = in2->data[i * stride_0_0] * in3->data[i * stride_1_0];
  }
}

//
// function [x,w,olPct] = ec_detr(x,w,ord,thr,itr)
void ec_detr(emxArray_real_T *cpu_x, emxArray_real_T *cpu_w,
             const real_T ord_data[], const int32_T ord_size[2],
             const real_T thr_data[], const int32_T[2], const real_T itr_data[],
             const int32_T[2], real_T olPct_data[], int32_T olPct_size[2])
{
  dim3 block;
  dim3 grid;
  emxArray_boolean_T d_gpu_x;
  emxArray_boolean_T *b_cpu_x;
  emxArray_creal_T b_gpu_d;
  emxArray_creal_T gpu_D;
  emxArray_creal_T gpu_V;
  emxArray_creal_T *cpu_D;
  emxArray_creal_T *cpu_V;
  emxArray_creal_T *cpu_d;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T *cpu_iv;
  emxArray_real_T b_gpu_V;
  emxArray_real_T b_gpu_r;
  emxArray_real_T b_gpu_regs;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_V;
  emxArray_real_T c_gpu_x;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_y;
  emxArray_real_T e_gpu_y;
  emxArray_real_T f_gpu_y;
  emxArray_real_T gpu_A;
  emxArray_real_T gpu_absdiff;
  emxArray_real_T gpu_d;
  emxArray_real_T gpu_dv;
  emxArray_real_T gpu_dv1;
  emxArray_real_T gpu_dv2;
  emxArray_real_T gpu_lin;
  emxArray_real_T gpu_r;
  emxArray_real_T gpu_r1;
  emxArray_real_T gpu_r2;
  emxArray_real_T gpu_r3;
  emxArray_real_T gpu_r4;
  emxArray_real_T gpu_regs;
  emxArray_real_T gpu_w;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_y;
  emxArray_real_T gpu_z;
  emxArray_real_T *b_cpu_V;
  emxArray_real_T *b_cpu_d;
  emxArray_real_T *b_cpu_r;
  emxArray_real_T *b_cpu_regs;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_V;
  emxArray_real_T *c_cpu_x;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_A;
  emxArray_real_T *cpu_absdiff;
  emxArray_real_T *cpu_dv;
  emxArray_real_T *cpu_dv1;
  emxArray_real_T *cpu_dv2;
  emxArray_real_T *cpu_lin;
  emxArray_real_T *cpu_r;
  emxArray_real_T *cpu_r1;
  emxArray_real_T *cpu_r2;
  emxArray_real_T *cpu_r3;
  emxArray_real_T *cpu_r4;
  emxArray_real_T *cpu_regs;
  emxArray_real_T *cpu_y;
  emxArray_real_T *cpu_z;
  emxArray_real_T *d_cpu_x;
  emxArray_real_T *d_cpu_y;
  emxArray_real_T *e_cpu_y;
  emxArray_real_T *f_cpu_y;
  emxArray_real_T *olPct;
  real_T g_cpu_y;
  real_T *g_gpu_y;
  int32_T b_i;
  int32_T i;
  int32_T i5;
  int32_T nFrames;
  boolean_T A_outdatedOnCpu;
  boolean_T b_x_outdatedOnCpu;
  boolean_T c_x_outdatedOnCpu;
  boolean_T d_outdatedOnCpu;
  boolean_T dv2_outdatedOnCpu;
  boolean_T dv2_outdatedOnGpu;
  boolean_T iv_outdatedOnGpu;
  boolean_T r1_outdatedOnCpu;
  boolean_T r1_outdatedOnGpu;
  boolean_T r_outdatedOnGpu;
  boolean_T w_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnCpu;
  boolean_T x_outdatedOnGpu;
  boolean_T y_outdatedOnCpu;
  boolean_T y_outdatedOnGpu;
  checkCudaError(mwCudaMalloc(&g_gpu_y, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real_T(&gpu_r4);
  gpuEmxReset_real_T(&d_gpu_y);
  gpuEmxReset_real_T(&gpu_r3);
  gpuEmxReset_real_T(&c_gpu_V);
  gpuEmxReset_real_T(&gpu_r2);
  gpuEmxReset_real_T(&b_gpu_regs);
  gpuEmxReset_real_T(&gpu_r1);
  gpuEmxReset_real_T(&gpu_r);
  gpuEmxReset_real_T(&gpu_absdiff);
  gpuEmxReset_real_T(&f_gpu_y);
  gpuEmxReset_real_T(&gpu_d);
  gpuEmxReset_real_T(&gpu_dv2);
  gpuEmxReset_real_T(&e_gpu_y);
  gpuEmxReset_real_T(&c_gpu_y);
  gpuEmxReset_real_T(&gpu_A);
  gpuEmxReset_real_T(&b_gpu_y);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_creal_T(&b_gpu_d);
  gpuEmxReset_real_T(&b_gpu_V);
  gpuEmxReset_creal_T(&gpu_D);
  gpuEmxReset_creal_T(&gpu_V);
  gpuEmxReset_real_T(&gpu_y);
  gpuEmxReset_real_T(&b_gpu_r);
  gpuEmxReset_real_T(&c_gpu_x);
  gpuEmxReset_real_T(&b_gpu_x);
  gpuEmxReset_boolean_T(&d_gpu_x);
  gpuEmxReset_real_T(&gpu_z);
  gpuEmxReset_real_T(&gpu_dv1);
  gpuEmxReset_real_T(&gpu_dv);
  gpuEmxReset_real_T(&gpu_lin);
  gpuEmxReset_real_T(&gpu_regs);
  gpuEmxReset_real_T(&gpu_w);
  gpuEmxReset_real_T(&gpu_x);
  r1_outdatedOnCpu = false;
  d_outdatedOnCpu = false;
  A_outdatedOnCpu = false;
  iv_outdatedOnGpu = false;
  y_outdatedOnGpu = false;
  c_x_outdatedOnCpu = false;
  b_x_outdatedOnCpu = false;
  dv2_outdatedOnCpu = false;
  w_outdatedOnGpu = true;
  x_outdatedOnCpu = false;
  x_outdatedOnGpu = true;
  y_outdatedOnCpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  Robust detrending - wrapper for GPU processing (CUDA source code)
  //
  //  INPUTS:
  //     see 'input validation' section below
  //  OUTPUTS:
  //     y = detrended data
  //     w = updated weights
  //     r = basis matrix used
  //
  //  This code was adapted from NoiseTools (audition.ens.fr/adc/NoiseTools)
  //
  //  Modified by: Kevin Tan, 2022 (kevmtan.github.io)
  //     - Added GPU support
  //     - Replaced bsxfun with implicit expansion operators (much faster)
  //     - Miscellaneous compute & readability enhancements
  //  Input validation
  // 'ec_detr:20' x (:,:){mustBeFloat}
  // 'ec_detr:21' w (:,:){mustBeFloat}
  // 'ec_detr:22' ord (1,:){mustBeFloat}
  // 'ec_detr:23' thr (1,:){mustBeFloat}
  // 'ec_detr:24' itr (1,:){mustBeFloat}
  //  Main
  // 'ec_detr:28' coder.gpu.kernelfun;
  //  trigger CUDA kernel generation
  // 'ec_detr:29' nFrames = size(x,1);
  nFrames = cpu_x->size[0];
  // 'ec_detr:30' nChs = size(x,2);
  // 'ec_detr:31' reps = numel(ord);
  // 'ec_detr:32' olPct = coder.nullcopy(nan(nChs,reps));
  emxInit_real_T(&olPct, 2, &uc_emlrtRTEI, true);
  i = olPct->size[0] * olPct->size[1];
  olPct->size[0] = cpu_x->size[1];
  olPct->size[1] = ord_size[1];
  emxEnsureCapacity_real_T(olPct, i, &ub_emlrtRTEI);
  //  Detrend by polynomial order
  // 'ec_detr:35' for ii = 1:reps
  b_i = ord_size[1];
  emxInit_real_T(&cpu_regs, 2, &vc_emlrtRTEI, true);
  emxInit_real_T(&cpu_lin, 2, &wc_emlrtRTEI, true);
  emxInit_real_T(&cpu_dv, 1, &jc_emlrtRTEI, true);
  emxInit_real_T(&cpu_dv1, 1, &jc_emlrtRTEI, true);
  emxInit_real_T(&cpu_z, 1, &xc_emlrtRTEI, true);
  emxInit_boolean_T(&b_cpu_x, 1, &bc_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_x, 1, &cc_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_x, 1, &cc_emlrtRTEI, true);
  emxInit_real_T(&cpu_r, 2, &yc_emlrtRTEI, true);
  emxInit_real_T(&cpu_y, 2, &ad_emlrtRTEI, true);
  emxInit_creal_T(&cpu_V, 2, &jc_emlrtRTEI, true);
  emxInit_creal_T(&cpu_D, 2, &jc_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_V, 2, &bd_emlrtRTEI, true);
  emxInit_creal_T(&cpu_d, 1, &cd_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &dd_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_y, 2, &ed_emlrtRTEI, true);
  emxInit_real_T(&cpu_A, 2, &nc_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_y, 2, &fd_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_y, 1, &gd_emlrtRTEI, true);
  emxInit_real_T(&cpu_dv2, 1, &hd_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_d, 1, &rc_emlrtRTEI, true);
  emxInit_real_T(&e_cpu_y, 1, &id_emlrtRTEI, true);
  emxInit_real_T(&cpu_absdiff, 1, &jd_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_r, 2, &kd_emlrtRTEI, true);
  emxInit_real_T(&cpu_r1, 2, &ld_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_regs, 2, &cc_emlrtRTEI, true);
  emxInit_real_T(&cpu_r2, 2, &fc_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_V, 2, &kc_emlrtRTEI, true);
  emxInit_real_T(&cpu_r3, 2, &fc_emlrtRTEI, true);
  emxInit_real_T(&f_cpu_y, 1, &pc_emlrtRTEI, true);
  emxInit_real_T(&cpu_r4, 2, &mc_emlrtRTEI, true);
  for (int32_T ii{0}; ii < b_i; ii++) {
    real_T d2scaled;
    int32_T i4;
    int32_T k;
    int32_T nx;
    int32_T nz;
    boolean_T regs_needsGpuEnsureCapacity;
    boolean_T regs_outdatedOnCpu;
    boolean_T validLaunchParams;
    // 'ec_detr:36' iOrd=ord(ii);
    // 'ec_detr:36' iThr=thr(ii);
    // 'ec_detr:36' iItr=itr(ii);
    //  copy
    // 'ec_detr:37' regs = regsFromBasis_lfn(nFrames,iOrd);
    // %%%%%%%%%%%%%%%%%%%%%%%%%% SUBFUNCTIONS
    // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //  Get regressors from polynomial function %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    // 'ec_detr:58' regs = coder.nullcopy(zeros(nFrames,iOrd,'like',iOrd));
    regs_outdatedOnCpu = false;
    i = cpu_regs->size[0] * cpu_regs->size[1];
    cpu_regs->size[0] = nFrames;
    cpu_regs->size[1] = static_cast<int32_T>(ord_data[ii]);
    emxEnsureCapacity_real_T(cpu_regs, i, &vb_emlrtRTEI);
    regs_needsGpuEnsureCapacity = true;
    // 'ec_detr:59' lin = linspace(-1,1,nFrames);
    i = cpu_lin->size[0] * cpu_lin->size[1];
    cpu_lin->size[0] = 1;
    cpu_lin->size[1] = nFrames;
    emxEnsureCapacity_real_T(cpu_lin, i, &wb_emlrtRTEI);
    dv2_outdatedOnGpu = true;
    if (nFrames >= 1) {
      gpuEmxEnsureCapacity_real_T(cpu_lin, &gpu_lin, true);
      dv2_outdatedOnGpu = false;
      ec_detr_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(nFrames,
                                                               gpu_lin);
      if (cpu_lin->size[1] >= 2) {
        ec_detr_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_lin);
        if (cpu_lin->size[1] >= 3) {
          d2scaled = 1.0 / (static_cast<real_T>(cpu_lin->size[1]) - 1.0);
          nz = cpu_lin->size[1];
          mwGetLaunchParameters1D(computeNumIters(nz - 3), &grid, &block,
                                  2147483647U);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel3<<<grid, block>>>(d2scaled, nz - 3, gpu_lin,
                                             cpu_lin->size[1U]);
          }
          if ((cpu_lin->size[1] & 1) == 1) {
            ec_detr_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                gpu_lin, cpu_lin->size[1U]);
          }
        }
      }
    }
    // 'ec_detr:60' for k = 1:iOrd
    nz = static_cast<int32_T>(ord_data[ii]);
    for (k = 0; k < nz; k++) {
      // 'ec_detr:61' regs(:,k) = lin.^k;
      nx = cpu_lin->size[1];
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      if (dv2_outdatedOnGpu) {
        gpuEmxEnsureCapacity_real_T(cpu_lin, &gpu_lin, true);
      }
      dv2_outdatedOnGpu = false;
      if (regs_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_real_T(cpu_regs, &gpu_regs, true);
      }
      regs_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_detr_kernel5<<<grid, block>>>(gpu_lin, k, nx - 1, gpu_regs,
                                         cpu_regs->size[0U]);
      }
      regs_outdatedOnCpu = true;
    }
    //  Get regressors from basis function
    //  Detrend per chan
    // 'ec_detr:40' for ch = 1:nChs
    i4 = cpu_x->size[1];
    if (i4 - 1 >= 0) {
      i5 = static_cast<int32_T>(itr_data[ii]);
    }
    for (int32_T ch{0}; ch < i4; ch++) {
      boolean_T dv1_needsGpuEnsureCapacity;
      boolean_T dv1_outdatedOnCpu;
      boolean_T dv_needsGpuEnsureCapacity;
      boolean_T dv_outdatedOnCpu;
      boolean_T z_outdatedOnCpu;
      // 'ec_detr:41' [x(:,ch),w(:,ch)] =
      // detrend_lfn(x(:,ch),w(:,ch),regs,iThr,iItr);
      i = cpu_dv->size[0];
      cpu_dv->size[0] = cpu_x->size[0];
      emxEnsureCapacity_real_T(cpu_dv, i, &xb_emlrtRTEI);
      nx = cpu_x->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      if (x_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
      dv_needsGpuEnsureCapacity = false;
      if (x_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_detr_kernel6<<<grid, block>>>(ch, gpu_x, nx, gpu_dv,
                                         cpu_x->size[0U]);
      }
      dv_outdatedOnCpu = true;
      i = cpu_dv1->size[0];
      cpu_dv1->size[0] = cpu_w->size[0];
      emxEnsureCapacity_real_T(cpu_dv1, i, &yb_emlrtRTEI);
      nx = cpu_w->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      if (y_outdatedOnCpu) {
        gpuEmxEnsureCapacity_real_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
      }
      gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
      dv1_needsGpuEnsureCapacity = false;
      if (w_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_w, cpu_w);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_detr_kernel7<<<grid, block>>>(ch, gpu_w, nx, gpu_dv1,
                                         cpu_w->size[0U]);
      }
      dv1_outdatedOnCpu = true;
      //  Standard detrending (trend fit to entire data)
      //  %%%%%%%%%%%%%%%%%%%%%%%%% The data are fit to the basis using weighted
      //  least squares. The weight is updated by setting samples for which the
      //  residual is greater than 'thresh' times its std to zero, and the fit
      //  is repeated at most 'niter'-1 times.
      //
      //  The choice of order (and basis) determines what complexity of the
      //  trend that can be removed.  It may be useful to first detrend with a
      //  low order to avoid fitting outliers, and then increase the order.
      //
      //  The tricky bit is to ensure that weighted means are removed before
      //  calculating the regression (see nt_regw)
      //  Detrend per polynomial order
      // 'ec_detr:80' z = coder.nullcopy(x);
      z_outdatedOnCpu = false;
      x_outdatedOnGpu = false;
      i = cpu_z->size[0];
      cpu_z->size[0] = cpu_x->size[0];
      emxEnsureCapacity_real_T(cpu_z, i, &ac_emlrtRTEI);
      x_needsGpuEnsureCapacity = true;
      // 'ec_detr:81' for ii = 1:iItr
      for (int32_T b_ii{0}; b_ii < i5; b_ii++) {
        real_T b_mn;
        real_T mn;
        //  Weighted regression
        // 'ec_detr:83' z = regw_lfn(x,w,r);
        //  Weighted Regression
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        //   b: regression matrix (apply to r to approximate x)
        //   z: regression (r*b)
        //  Discard dimensions of r with eigenvalue lower than this
        // 'ec_detr:102' thrPCA = 1e-7;
        // zeros(1,1,class(x));
        //  Save weighted mean
        // 'ec_detr:105' mn = x - demean_lfn(x,w);
        //  Demean
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        // 'ec_detr:124' mn = sum(x.*w,1) ./ (sum(w,1)+eps);
        if (cpu_dv->size[0] == cpu_dv1->size[0]) {
          i = c_cpu_x->size[0];
          c_cpu_x->size[0] = cpu_dv->size[0];
          emxEnsureCapacity_real_T(c_cpu_x, i, &cc_emlrtRTEI);
          nx = cpu_dv->size[0] - 1;
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          if (dv_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          }
          dv_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(c_cpu_x, &b_gpu_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel8<<<grid, block>>>(gpu_dv1, gpu_dv, nx, b_gpu_x);
          }
          b_x_outdatedOnCpu = true;
        } else {
          if (b_x_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(c_cpu_x, &b_gpu_x);
          }
          if (dv_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
          }
          dv_outdatedOnCpu = false;
          gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          dv1_outdatedOnCpu = false;
          times(c_cpu_x, cpu_dv, cpu_dv1);
          b_x_outdatedOnCpu = false;
          dv1_needsGpuEnsureCapacity = true;
          dv_needsGpuEnsureCapacity = true;
        }
        nx = c_cpu_x->size[0];
        if (c_cpu_x->size[0] == 0) {
          g_cpu_y = 0.0;
        } else {
          if (b_x_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(c_cpu_x, &b_gpu_x);
          }
          b_x_outdatedOnCpu = false;
          g_cpu_y = c_cpu_x->data[0];
          for (k = 0; k <= nx - 2; k++) {
            g_cpu_y += c_cpu_x->data[k + 1];
          }
        }
        nx = cpu_dv1->size[0];
        if (cpu_dv1->size[0] == 0) {
          d2scaled = 0.0;
        } else {
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          dv1_outdatedOnCpu = false;
          d2scaled = cpu_dv1->data[0];
          dv1_needsGpuEnsureCapacity = true;
          for (k = 0; k <= nx - 2; k++) {
            d2scaled += cpu_dv1->data[k + 1];
          }
        }
        mn = g_cpu_y / (d2scaled + 2.2204460492503131E-16);
        // 'ec_detr:125' x = x - mn;
        //  Fit weighted regression
        // 'ec_detr:108' x = demean_lfn(x,w).* w;
        //  Demean
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        // 'ec_detr:124' mn = sum(x.*w,1) ./ (sum(w,1)+eps);
        if (cpu_dv->size[0] == cpu_dv1->size[0]) {
          i = d_cpu_x->size[0];
          d_cpu_x->size[0] = cpu_dv->size[0];
          emxEnsureCapacity_real_T(d_cpu_x, i, &cc_emlrtRTEI);
          nx = cpu_dv->size[0] - 1;
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          if (dv1_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
          }
          dv1_needsGpuEnsureCapacity = false;
          if (dv_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          }
          dv_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(d_cpu_x, &c_gpu_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel9<<<grid, block>>>(gpu_dv1, gpu_dv, nx, c_gpu_x);
          }
          c_x_outdatedOnCpu = true;
        } else {
          if (c_x_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(d_cpu_x, &c_gpu_x);
          }
          if (dv_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
          }
          dv_outdatedOnCpu = false;
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          dv1_outdatedOnCpu = false;
          times(d_cpu_x, cpu_dv, cpu_dv1);
          c_x_outdatedOnCpu = false;
          dv1_needsGpuEnsureCapacity = true;
          dv_needsGpuEnsureCapacity = true;
        }
        nx = d_cpu_x->size[0];
        if (d_cpu_x->size[0] == 0) {
          g_cpu_y = 0.0;
        } else {
          if (c_x_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(d_cpu_x, &c_gpu_x);
          }
          c_x_outdatedOnCpu = false;
          g_cpu_y = d_cpu_x->data[0];
          for (k = 0; k <= nx - 2; k++) {
            g_cpu_y += d_cpu_x->data[k + 1];
          }
        }
        nx = cpu_dv1->size[0];
        if (cpu_dv1->size[0] == 0) {
          d2scaled = 0.0;
        } else {
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          dv1_outdatedOnCpu = false;
          d2scaled = cpu_dv1->data[0];
          dv1_needsGpuEnsureCapacity = true;
          for (k = 0; k <= nx - 2; k++) {
            d2scaled += cpu_dv1->data[k + 1];
          }
        }
        b_mn = g_cpu_y / (d2scaled + 2.2204460492503131E-16);
        // 'ec_detr:125' x = x - mn;
        // 'ec_detr:109' r = demean_lfn(r,w);
        //  Demean
        //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
        // 'ec_detr:124' mn = sum(x.*w,1) ./ (sum(w,1)+eps);
        nx = cpu_dv1->size[0];
        if (cpu_dv1->size[0] == 0) {
          g_cpu_y = 0.0;
        } else {
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          dv1_outdatedOnCpu = false;
          g_cpu_y = cpu_dv1->data[0];
          dv1_needsGpuEnsureCapacity = true;
          for (k = 0; k <= nx - 2; k++) {
            g_cpu_y += cpu_dv1->data[k + 1];
          }
        }
        // 'ec_detr:125' x = x - mn;
        if (cpu_regs->size[0] == cpu_dv1->size[0]) {
          i = b_cpu_regs->size[0] * b_cpu_regs->size[1];
          b_cpu_regs->size[0] = cpu_regs->size[0];
          b_cpu_regs->size[1] = cpu_regs->size[1];
          emxEnsureCapacity_real_T(b_cpu_regs, i, &cc_emlrtRTEI);
          nx = cpu_regs->size[1] - 1;
          nz = cpu_regs->size[0] - 1;
          mwGetLaunchParameters1D(computeNumIters(nx, nz), &grid, &block,
                                  2147483647U);
          if (dv1_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
          }
          dv1_needsGpuEnsureCapacity = false;
          if (regs_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_regs, &gpu_regs, true);
          }
          regs_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(b_cpu_regs, &b_gpu_regs, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel10<<<grid, block>>>(gpu_dv1, gpu_regs, nz, nx,
                                              b_gpu_regs, b_cpu_regs->size[0U],
                                              cpu_regs->size[0U]);
          }
          gpuEmxMemcpyGpuToCpu_real_T(b_cpu_regs, &b_gpu_regs);
          coder::sum(b_cpu_regs, b_cpu_r);
        } else {
          if (regs_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_regs, &gpu_regs);
          }
          regs_outdatedOnCpu = false;
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          dv1_outdatedOnCpu = false;
          binary_expand_op_4(b_cpu_r, cpu_regs, cpu_dv1);
          dv1_needsGpuEnsureCapacity = true;
          regs_needsGpuEnsureCapacity = true;
        }
        g_cpu_y += 2.2204460492503131E-16;
        i = cpu_r->size[0] * cpu_r->size[1];
        cpu_r->size[0] = cpu_regs->size[0];
        cpu_r->size[1] = cpu_regs->size[1];
        emxEnsureCapacity_real_T(cpu_r, i, &dc_emlrtRTEI);
        nx = cpu_regs->size[1] - 1;
        nz = cpu_regs->size[0] - 1;
        mwGetLaunchParameters1D(computeNumIters(nx, nz), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(b_cpu_r, &gpu_r, false);
        if (regs_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_real_T(cpu_regs, &gpu_regs, true);
        }
        regs_needsGpuEnsureCapacity = false;
        gpuEmxEnsureCapacity_real_T(cpu_r, &b_gpu_r, true);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, b_cpu_r);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel11<<<grid, block>>>(g_cpu_y, gpu_r, gpu_regs, nz, nx,
                                            b_gpu_r, cpu_r->size[0U],
                                            cpu_regs->size[0U]);
        }
        r_outdatedOnGpu = false;
        //  remove channel-specific-weighted mean from regressor
        // 'ec_detr:110' rr = r.*w;
        if (cpu_r->size[0] == cpu_dv1->size[0]) {
          i = cpu_r1->size[0] * cpu_r1->size[1];
          cpu_r1->size[0] = cpu_r->size[0];
          cpu_r1->size[1] = cpu_r->size[1];
          emxEnsureCapacity_real_T(cpu_r1, i, &ec_emlrtRTEI);
          nx = cpu_r->size[1] - 1;
          nz = cpu_r->size[0] - 1;
          mwGetLaunchParameters1D(computeNumIters(nx, nz), &grid, &block,
                                  2147483647U);
          if (dv1_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
          }
          dv1_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(cpu_r1, &gpu_r1, true);
          y_outdatedOnCpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel12<<<grid, block>>>(gpu_dv1, b_gpu_r, nz, nx, gpu_r1,
                                              cpu_r1->size[0U],
                                              cpu_r->size[0U]);
          }
          r1_outdatedOnGpu = false;
          r1_outdatedOnCpu = true;
        } else {
          if (r1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_r1, &gpu_r1);
          }
          gpuEmxMemcpyGpuToCpu_real_T(cpu_r, &b_gpu_r);
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          dv1_outdatedOnCpu = false;
          binary_expand_op_3(cpu_r1, cpu_r, cpu_dv1);
          r1_outdatedOnCpu = false;
          r1_outdatedOnGpu = true;
          dv1_needsGpuEnsureCapacity = true;
          y_outdatedOnCpu = true;
        }
        // 'ec_detr:111' [V,D] = eig(rr'*rr);
        i = cpu_r2->size[0] * cpu_r2->size[1];
        cpu_r2->size[0] = cpu_r1->size[1];
        cpu_r2->size[1] = cpu_r1->size[0];
        emxEnsureCapacity_real_T(cpu_r2, i, &fc_emlrtRTEI);
        nx = cpu_r1->size[0] - 1;
        nz = cpu_r1->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(nx, nz), &grid, &block,
                                2147483647U);
        if (y_outdatedOnCpu) {
          gpuEmxEnsureCapacity_real_T(cpu_r1, &gpu_r1, !r1_outdatedOnGpu);
        }
        gpuEmxEnsureCapacity_real_T(cpu_r2, &gpu_r2, true);
        if (r1_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_r1, cpu_r1);
        }
        r1_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel13<<<grid, block>>>(gpu_r1, nz, nx, gpu_r2,
                                            cpu_r2->size[0U], cpu_r1->size[0U]);
        }
        x_outdatedOnGpu = false;
        coder::internal::blas::mtimes(cpu_r2, &gpu_r2, &x_outdatedOnGpu, cpu_r1,
                                      &gpu_r1, &r1_outdatedOnGpu, cpu_y,
                                      &y_outdatedOnCpu, &gpu_y,
                                      &y_outdatedOnGpu);
        if (y_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &gpu_y);
        }
        coder::eig(cpu_y, cpu_V, cpu_D);
        // 'ec_detr:112' V = real(V);
        i = b_cpu_V->size[0] * b_cpu_V->size[1];
        b_cpu_V->size[0] = cpu_V->size[0];
        b_cpu_V->size[1] = cpu_V->size[1];
        emxEnsureCapacity_real_T(b_cpu_V, i, &gc_emlrtRTEI);
        nx = cpu_V->size[0] * cpu_V->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_creal_T(cpu_V, &gpu_V, false);
        gpuEmxEnsureCapacity_real_T(b_cpu_V, &b_gpu_V, true);
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_V, cpu_V);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel14<<<grid, block>>>(gpu_V, nx, b_gpu_V);
        }
        // 'ec_detr:113' D = real(diag(D));
        if ((cpu_D->size[0] == 1) && (cpu_D->size[1] == 1)) {
          i = cpu_d->size[0];
          cpu_d->size[0] = 1;
          emxEnsureCapacity_creal_T(cpu_d, i, &ic_emlrtRTEI);
          cpu_d->data[0] = cpu_D->data[0];
          dv2_outdatedOnCpu = false;
        } else {
          nz = cpu_D->size[0];
          nx = cpu_D->size[1];
          if (nz <= nx) {
            nx = nz;
          }
          if (cpu_D->size[1] > 0) {
            nz = nx;
          } else {
            nz = 0;
          }
          i = cpu_d->size[0];
          cpu_d->size[0] = nz;
          emxEnsureCapacity_creal_T(cpu_d, i, &hc_emlrtRTEI);
          nz--;
          mwGetLaunchParameters1D(computeNumIters(nz), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_creal_T(cpu_D, &gpu_D, false);
          gpuEmxEnsureCapacity_creal_T(cpu_d, &b_gpu_d, true);
          gpuEmxMemcpyCpuToGpu_creal_T(&gpu_D, cpu_D);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel27<<<grid, block>>>(gpu_D, nz, b_gpu_d,
                                              cpu_D->size[0U]);
          }
          dv2_outdatedOnCpu = true;
        }
        // 'ec_detr:114' V = V(:,D/max(D)>thrPCA);
        nx = cpu_d->size[0];
        if (dv2_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_creal_T(cpu_d, &b_gpu_d);
        }
        d2scaled = cpu_d->data[0].re;
        for (i = 0; i <= nx - 2; i++) {
          if (std::isnan(cpu_d->data[i + 1].re)) {
            dv2_outdatedOnCpu = false;
          } else if (std::isnan(d2scaled)) {
            dv2_outdatedOnCpu = true;
          } else {
            dv2_outdatedOnCpu = (d2scaled < cpu_d->data[i + 1].re);
          }
          if (dv2_outdatedOnCpu) {
            d2scaled = cpu_d->data[i + 1].re;
          }
        }
        k = cpu_d->size[0] - 1;
        nz = 0;
        for (i = 0; i <= k; i++) {
          if (cpu_d->data[i].re / d2scaled > 1.0E-7) {
            nz++;
          }
        }
        i = cpu_iv->size[0];
        cpu_iv->size[0] = nz;
        emxEnsureCapacity_int32_T(cpu_iv, i, &jc_emlrtRTEI);
        nx = 0;
        for (i = 0; i <= k; i++) {
          if (cpu_d->data[i].re / d2scaled > 1.0E-7) {
            cpu_iv->data[nx] = i;
            iv_outdatedOnGpu = true;
            nx++;
          }
        }
        nz = b_cpu_V->size[0];
        i = c_cpu_V->size[0] * c_cpu_V->size[1];
        c_cpu_V->size[0] = b_cpu_V->size[0];
        c_cpu_V->size[1] = cpu_iv->size[0];
        emxEnsureCapacity_real_T(c_cpu_V, i, &kc_emlrtRTEI);
        nx = cpu_iv->size[0] - 1;
        mwGetLaunchParameters1D(computeNumIters(nx, nz - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, !iv_outdatedOnGpu);
        gpuEmxEnsureCapacity_real_T(c_cpu_V, &c_gpu_V, true);
        if (iv_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_int32_T(&gpu_iv, cpu_iv);
        }
        iv_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel15<<<grid, block>>>(gpu_iv, b_gpu_V, nz - 1, nx,
                                            c_gpu_V, c_cpu_V->size[0U],
                                            b_cpu_V->size[0U]);
        }
        i = b_cpu_V->size[0] * b_cpu_V->size[1];
        b_cpu_V->size[0] = c_cpu_V->size[0];
        b_cpu_V->size[1] = c_cpu_V->size[1];
        emxEnsureCapacity_real_T(b_cpu_V, i, &lc_emlrtRTEI);
        nx = c_cpu_V->size[0] * c_cpu_V->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(b_cpu_V, &b_gpu_V, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel16<<<grid, block>>>(c_gpu_V, nx, b_gpu_V);
        }
        dv2_outdatedOnGpu = false;
        //  discard weak dims
        // 'ec_detr:115' rr = rr*V;
        x_needsGpuEnsureCapacity = false;
        i = cpu_r4->size[0] * cpu_r4->size[1];
        cpu_r4->size[0] = cpu_r1->size[0];
        cpu_r4->size[1] = cpu_r1->size[1];
        emxEnsureCapacity_real_T(cpu_r4, i, &mc_emlrtRTEI);
        nz = cpu_r1->size[0] * cpu_r1->size[1] - 1;
        for (i = 0; i <= nz; i++) {
          if (r1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_r1, &gpu_r1);
          }
          r1_outdatedOnCpu = false;
          cpu_r4->data[i] = cpu_r1->data[i];
          x_needsGpuEnsureCapacity = true;
        }
        coder::internal::blas::mtimes(
            cpu_r4, &gpu_r4, &x_needsGpuEnsureCapacity, b_cpu_V, &b_gpu_V,
            &dv2_outdatedOnGpu, cpu_r1, &r1_outdatedOnCpu, &gpu_r1,
            &r1_outdatedOnGpu);
        y_outdatedOnCpu = true;
        // 'ec_detr:116' b = (x'*rr) / (rr'*rr);
        if (cpu_dv->size[0] == cpu_dv1->size[0]) {
          i = cpu_A->size[0] * cpu_A->size[1];
          cpu_A->size[0] = 1;
          cpu_A->size[1] = cpu_dv->size[0];
          emxEnsureCapacity_real_T(cpu_A, i, &nc_emlrtRTEI);
          nx = cpu_dv->size[0] - 1;
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          if (dv1_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
          }
          dv1_needsGpuEnsureCapacity = false;
          if (dv_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          }
          dv_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(cpu_A, &gpu_A, true);
          x_needsGpuEnsureCapacity = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel17<<<grid, block>>>(gpu_dv1, b_mn, gpu_dv, nx, gpu_A);
          }
          x_outdatedOnGpu = false;
          A_outdatedOnCpu = true;
        } else {
          if (A_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_A, &gpu_A);
          }
          if (dv_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
          }
          dv_outdatedOnCpu = false;
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          dv1_outdatedOnCpu = false;
          binary_expand_op_2(cpu_A, cpu_dv, b_mn, cpu_dv1);
          A_outdatedOnCpu = false;
          x_outdatedOnGpu = true;
          dv1_needsGpuEnsureCapacity = true;
          dv_needsGpuEnsureCapacity = true;
          x_needsGpuEnsureCapacity = true;
        }
        if ((cpu_A->size[1] == 0) || (cpu_r1->size[0] == 0) ||
            (cpu_r1->size[1] == 0)) {
          i = b_cpu_y->size[0] * b_cpu_y->size[1];
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = cpu_r1->size[1];
          emxEnsureCapacity_real_T(b_cpu_y, i, &oc_emlrtRTEI);
          nz = cpu_r1->size[1];
          mwGetLaunchParameters1D(computeNumIters(nz - 1), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel18<<<grid, block>>>(nz - 1, b_gpu_y);
          }
          w_outdatedOnGpu = false;
          dv2_outdatedOnCpu = true;
        } else {
          i = b_cpu_y->size[0] * b_cpu_y->size[1];
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = cpu_r1->size[1];
          emxEnsureCapacity_real_T(b_cpu_y, i, &c_emlrtRTEI);
          d2scaled = 1.0;
          g_cpu_y = 0.0;
          if (x_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_A, &gpu_A, !x_outdatedOnGpu);
          }
          gpuEmxEnsureCapacity_real_T(cpu_r1, &gpu_r1, !r1_outdatedOnGpu);
          y_outdatedOnCpu = false;
          gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
          if (x_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_A, cpu_A);
          }
          if (r1_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_r1, cpu_r1);
          }
          r1_outdatedOnGpu = false;
          cublasCheck(
              hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N, 1,
                          cpu_r1->size[1], cpu_A->size[1], (double *)&d2scaled,
                          (double *)&gpu_A.data[0], 1,
                          (double *)&gpu_r1.data[0], cpu_A->size[1],
                          (double *)&g_cpu_y, (double *)&b_gpu_y.data[0], 1),
              __FILE__, __LINE__);
          w_outdatedOnGpu = false;
          dv2_outdatedOnCpu = true;
        }
        i = cpu_r3->size[0] * cpu_r3->size[1];
        cpu_r3->size[0] = cpu_r1->size[1];
        cpu_r3->size[1] = cpu_r1->size[0];
        emxEnsureCapacity_real_T(cpu_r3, i, &fc_emlrtRTEI);
        nx = cpu_r1->size[0] - 1;
        nz = cpu_r1->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(nx, nz), &grid, &block,
                                2147483647U);
        if (y_outdatedOnCpu) {
          gpuEmxEnsureCapacity_real_T(cpu_r1, &gpu_r1, !r1_outdatedOnGpu);
        }
        gpuEmxEnsureCapacity_real_T(cpu_r3, &gpu_r3, true);
        if (r1_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_r1, cpu_r1);
        }
        r1_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel19<<<grid, block>>>(gpu_r1, nz, nx, gpu_r3,
                                            cpu_r3->size[0U], cpu_r1->size[0U]);
        }
        x_outdatedOnGpu = false;
        coder::internal::blas::mtimes(cpu_r3, &gpu_r3, &x_outdatedOnGpu, cpu_r1,
                                      &gpu_r1, &r1_outdatedOnGpu, c_cpu_y,
                                      &y_outdatedOnCpu, &c_gpu_y,
                                      &y_outdatedOnGpu);
        if (y_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(c_cpu_y, &c_gpu_y);
        }
        y_outdatedOnCpu = false;
        coder::internal::mrdiv(b_cpu_y, &dv2_outdatedOnCpu, &b_gpu_y,
                               &w_outdatedOnGpu, c_cpu_y);
        // 'ec_detr:117' z = r*(V*b') + mn;
        i = f_cpu_y->size[0];
        f_cpu_y->size[0] = b_cpu_y->size[1];
        emxEnsureCapacity_real_T(f_cpu_y, i, &pc_emlrtRTEI);
        nx = b_cpu_y->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, !w_outdatedOnGpu);
        gpuEmxEnsureCapacity_real_T(f_cpu_y, &d_gpu_y, true);
        if (w_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&b_gpu_y, b_cpu_y);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel20<<<grid, block>>>(b_gpu_y, nx, d_gpu_y);
        }
        w_outdatedOnGpu = false;
        dv2_outdatedOnCpu = true;
        coder::internal::blas::b_mtimes(b_cpu_V, &b_gpu_V, &dv2_outdatedOnGpu,
                                        f_cpu_y, &d_gpu_y, &w_outdatedOnGpu,
                                        d_cpu_y, &y_outdatedOnCpu, &e_gpu_y,
                                        &y_outdatedOnGpu);
        coder::internal::blas::b_mtimes(cpu_r, &b_gpu_r, &r_outdatedOnGpu,
                                        d_cpu_y, &e_gpu_y, &y_outdatedOnGpu,
                                        cpu_dv2, &dv2_outdatedOnCpu, &gpu_dv2,
                                        &dv2_outdatedOnGpu);
        if (cpu_dv2->size[0] == cpu_dv->size[0]) {
          i = cpu_z->size[0];
          cpu_z->size[0] = cpu_dv2->size[0];
          emxEnsureCapacity_real_T(cpu_z, i, &qc_emlrtRTEI);
          nx = cpu_dv2->size[0] - 1;
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          if (dv_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          }
          dv_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(cpu_dv2, &gpu_dv2, !dv2_outdatedOnGpu);
          gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, true);
          x_needsGpuEnsureCapacity = false;
          if (dv2_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_dv2, cpu_dv2);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel21<<<grid, block>>>(mn, gpu_dv, gpu_dv2, nx, gpu_z);
          }
          x_outdatedOnGpu = false;
          z_outdatedOnCpu = true;
        } else {
          if (z_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_z, &gpu_z);
          }
          if (dv2_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv2, &gpu_dv2);
          }
          if (dv_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
          }
          dv_outdatedOnCpu = false;
          binary_expand_op_1(cpu_z, cpu_dv2, cpu_dv, mn);
          z_outdatedOnCpu = false;
          x_outdatedOnGpu = true;
          dv_needsGpuEnsureCapacity = true;
          x_needsGpuEnsureCapacity = true;
        }
        //  Find outliers
        // 'ec_detr:86' d = (x-z).*w;
        if (cpu_dv->size[0] == 1) {
          nx = cpu_z->size[0];
        } else {
          nx = cpu_dv->size[0];
        }
        if ((cpu_dv->size[0] == cpu_z->size[0]) && (nx == cpu_dv1->size[0])) {
          i = b_cpu_d->size[0];
          b_cpu_d->size[0] = cpu_dv->size[0];
          emxEnsureCapacity_real_T(b_cpu_d, i, &rc_emlrtRTEI);
          nx = cpu_dv->size[0] - 1;
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          if (dv1_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
          }
          dv1_needsGpuEnsureCapacity = false;
          if (x_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, !x_outdatedOnGpu);
          }
          x_needsGpuEnsureCapacity = false;
          if (dv_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
          }
          dv_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_real_T(b_cpu_d, &gpu_d, true);
          dv2_outdatedOnGpu = false;
          if (x_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_z, cpu_z);
          }
          x_outdatedOnGpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel22<<<grid, block>>>(gpu_dv1, gpu_z, gpu_dv, nx,
                                              gpu_d);
          }
          dv2_outdatedOnCpu = false;
          d_outdatedOnCpu = true;
        } else {
          if (d_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(b_cpu_d, &gpu_d);
          }
          if (dv_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv, &gpu_dv);
          }
          dv_outdatedOnCpu = false;
          if (z_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_z, &gpu_z);
          }
          z_outdatedOnCpu = false;
          if (dv1_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(cpu_dv1, &gpu_dv1);
          }
          binary_expand_op(b_cpu_d, cpu_dv, cpu_z, cpu_dv1);
          d_outdatedOnCpu = false;
          dv2_outdatedOnCpu = true;
          dv1_needsGpuEnsureCapacity = true;
          x_needsGpuEnsureCapacity = true;
          dv_needsGpuEnsureCapacity = true;
          dv2_outdatedOnGpu = true;
        }
        // 'ec_detr:87' thrItr = iThr*std(d);
        nx = b_cpu_d->size[0] - 1;
        if (b_cpu_d->size[0] == 0) {
          g_cpu_y = rtNaN;
        } else if (b_cpu_d->size[0] == 1) {
          if (dv2_outdatedOnGpu) {
            gpuEmxEnsureCapacity_real_T(b_cpu_d, &gpu_d, !dv2_outdatedOnCpu);
          }
          dv2_outdatedOnGpu = false;
          if (dv2_outdatedOnCpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_d, b_cpu_d);
          }
          dv2_outdatedOnCpu = false;
          ec_detr_kernel25<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_d,
                                                                    g_gpu_y);
          checkCudaError(
              hipMemcpy(&g_cpu_y, g_gpu_y, 8UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        } else {
          if (d_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real_T(b_cpu_d, &gpu_d);
          }
          d_outdatedOnCpu = false;
          d2scaled = b_cpu_d->data[0];
          for (k = 0; k < nx; k++) {
            d2scaled += b_cpu_d->data[k + 1];
          }
          d2scaled /= static_cast<real_T>(b_cpu_d->size[0]);
          i = cpu_absdiff->size[0];
          cpu_absdiff->size[0] = b_cpu_d->size[0];
          emxEnsureCapacity_real_T(cpu_absdiff, i, &sc_emlrtRTEI);
          mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real_T(b_cpu_d, &gpu_d, !dv2_outdatedOnCpu);
          dv2_outdatedOnGpu = false;
          gpuEmxEnsureCapacity_real_T(cpu_absdiff, &gpu_absdiff, true);
          if (dv2_outdatedOnCpu) {
            gpuEmxMemcpyCpuToGpu_real_T(&gpu_d, b_cpu_d);
          }
          dv2_outdatedOnCpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_detr_kernel26<<<grid, block>>>(d2scaled, gpu_d, nx, gpu_absdiff);
          }
          cublasCheck(hipblasDnrm2(getCublasGlobalHandle(), b_cpu_d->size[0],
                                  (double *)&gpu_absdiff.data[0], 1,
                                  (double *)&g_cpu_y),
                      __FILE__, __LINE__);
          g_cpu_y /= std::sqrt(static_cast<real_T>(b_cpu_d->size[0]) - 1.0);
        }
        // 'ec_detr:88' w(abs(d)>thrItr) = 0;
        nx = b_cpu_d->size[0];
        i = e_cpu_y->size[0];
        e_cpu_y->size[0] = b_cpu_d->size[0];
        emxEnsureCapacity_real_T(e_cpu_y, i, &tc_emlrtRTEI);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        if (dv2_outdatedOnGpu) {
          gpuEmxEnsureCapacity_real_T(b_cpu_d, &gpu_d, !dv2_outdatedOnCpu);
        }
        gpuEmxEnsureCapacity_real_T(e_cpu_y, &f_gpu_y, true);
        if (dv2_outdatedOnCpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_d, b_cpu_d);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel23<<<grid, block>>>(gpu_d, nx - 1, f_gpu_y);
        }
        k = e_cpu_y->size[0] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        if (dv1_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
        }
        dv1_needsGpuEnsureCapacity = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel24<<<grid, block>>>(thr_data[ii] * g_cpu_y, f_gpu_y, k,
                                            gpu_dv1);
        }
        dv1_outdatedOnCpu = true;
        //  update weights
      }
      //  Remove trends
      // 'ec_detr:92' x = x-z;
      nz = cpu_x->size[0];
      if (nz == cpu_z->size[0]) {
        mwGetLaunchParameters1D(computeNumIters(nz - 1), &grid, &block,
                                2147483647U);
        if (x_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_real_T(cpu_z, &gpu_z, !x_outdatedOnGpu);
        }
        x_needsGpuEnsureCapacity = false;
        if (x_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_z, cpu_z);
        }
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_detr_kernel28<<<grid, block>>>(gpu_z, ch, nz - 1, gpu_x,
                                            cpu_x->size[0U]);
        }
        x_outdatedOnGpu = false;
        x_outdatedOnCpu = true;
      } else {
        if (x_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_x, &gpu_x);
        }
        if (z_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_z, &gpu_z);
        }
        binary_expand_op_5(cpu_x, ch, cpu_z);
        x_outdatedOnCpu = false;
        x_outdatedOnGpu = true;
        x_needsGpuEnsureCapacity = true;
      }
      nz = cpu_w->size[0];
      mwGetLaunchParameters1D(computeNumIters(nz - 1), &grid, &block,
                              2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_detr_kernel29<<<grid, block>>>(gpu_dv1, ch, nz - 1, gpu_w,
                                          cpu_w->size[0U]);
      }
      dv2_outdatedOnCpu = true;
      //  Detrend
      // 'ec_detr:42' olPct(ch,ii) = sum(~w(:,ch));
      i = b_cpu_x->size[0];
      b_cpu_x->size[0] = cpu_w->size[0];
      emxEnsureCapacity_boolean_T(b_cpu_x, i, &bc_emlrtRTEI);
      nx = cpu_w->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(nx), &grid, &block, 2147483647U);
      y_outdatedOnCpu = false;
      gpuEmxEnsureCapacity_boolean_T(b_cpu_x, &d_gpu_x, true);
      w_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_detr_kernel30<<<grid, block>>>(ch, gpu_w, nx, d_gpu_x,
                                          cpu_w->size[0U]);
      }
      nx = b_cpu_x->size[0];
      if (b_cpu_x->size[0] == 0) {
        nz = 0;
      } else {
        gpuEmxMemcpyGpuToCpu_boolean_T(b_cpu_x, &d_gpu_x);
        nz = b_cpu_x->data[0];
        for (k = 0; k <= nx - 2; k++) {
          nz += b_cpu_x->data[k + 1];
        }
      }
      olPct->data[ch + olPct->size[0] * ii] = nz;
      //  Outlier percentages
    }
  }
  emxFree_real_T(&cpu_r4);
  emxFree_real_T(&f_cpu_y);
  emxFree_real_T(&cpu_r3);
  emxFree_real_T(&c_cpu_V);
  emxFree_real_T(&cpu_r2);
  emxFree_real_T(&b_cpu_regs);
  emxFree_real_T(&cpu_r1);
  emxFree_real_T(&b_cpu_r);
  emxFree_real_T(&cpu_absdiff);
  emxFree_real_T(&e_cpu_y);
  emxFree_real_T(&b_cpu_d);
  emxFree_real_T(&cpu_dv2);
  emxFree_real_T(&d_cpu_y);
  emxFree_real_T(&c_cpu_y);
  emxFree_real_T(&cpu_A);
  emxFree_real_T(&b_cpu_y);
  emxFree_int32_T(&cpu_iv);
  emxFree_creal_T(&cpu_d);
  emxFree_real_T(&b_cpu_V);
  emxFree_creal_T(&cpu_D);
  emxFree_creal_T(&cpu_V);
  emxFree_real_T(&cpu_y);
  emxFree_real_T(&cpu_r);
  emxFree_real_T(&d_cpu_x);
  emxFree_real_T(&c_cpu_x);
  emxFree_boolean_T(&b_cpu_x);
  emxFree_real_T(&cpu_z);
  emxFree_real_T(&cpu_dv1);
  emxFree_real_T(&cpu_dv);
  emxFree_real_T(&cpu_lin);
  emxFree_real_T(&cpu_regs);
  //  Sum outliers across chans
  // 'ec_detr:47' olPct = sum(olPct,1,"omitnan");
  coder::sum(olPct, olPct_data, olPct_size);
  emxFree_real_T(&olPct);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (x_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_real_T(cpu_x, &gpu_x);
  }
  if (dv2_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_real_T(cpu_w, &gpu_w);
  }
  gpuEmxFree_real_T(&gpu_x);
  gpuEmxFree_real_T(&gpu_w);
  gpuEmxFree_real_T(&gpu_regs);
  gpuEmxFree_real_T(&gpu_lin);
  gpuEmxFree_real_T(&gpu_dv);
  gpuEmxFree_real_T(&gpu_dv1);
  gpuEmxFree_real_T(&gpu_z);
  gpuEmxFree_boolean_T(&d_gpu_x);
  gpuEmxFree_real_T(&b_gpu_x);
  gpuEmxFree_real_T(&c_gpu_x);
  gpuEmxFree_real_T(&b_gpu_r);
  gpuEmxFree_real_T(&gpu_y);
  gpuEmxFree_creal_T(&gpu_V);
  gpuEmxFree_creal_T(&gpu_D);
  gpuEmxFree_real_T(&b_gpu_V);
  gpuEmxFree_creal_T(&b_gpu_d);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_real_T(&b_gpu_y);
  gpuEmxFree_real_T(&gpu_A);
  gpuEmxFree_real_T(&c_gpu_y);
  gpuEmxFree_real_T(&e_gpu_y);
  gpuEmxFree_real_T(&gpu_dv2);
  gpuEmxFree_real_T(&gpu_d);
  gpuEmxFree_real_T(&f_gpu_y);
  gpuEmxFree_real_T(&gpu_absdiff);
  gpuEmxFree_real_T(&gpu_r);
  gpuEmxFree_real_T(&gpu_r1);
  gpuEmxFree_real_T(&b_gpu_regs);
  gpuEmxFree_real_T(&gpu_r2);
  gpuEmxFree_real_T(&c_gpu_V);
  gpuEmxFree_real_T(&gpu_r3);
  gpuEmxFree_real_T(&d_gpu_y);
  gpuEmxFree_real_T(&gpu_r4);
  checkCudaError(mwCudaFree(g_gpu_y), __FILE__, __LINE__);
}

// End of code generation (ec_detr.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mtimes.cu
//
// Code generation for function 'mtimes'
//

// Include files
#include "mtimes.h"
#include "ec_detr_fp_data.h"
#include "ec_detr_fp_emxutil.h"
#include "ec_detr_fp_mexutil.h"
#include "ec_detr_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Variable Definitions
static emlrtRTEInfo
    d_emlrtRTEI{
        140,      // lineNo
        5,        // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

// Function Definitions
//
//
namespace coder {
namespace internal {
namespace blas {
void b_mtimes(emxArray_real_T *cpu_A, emxArray_real_T *gpu_A,
              boolean_T *A_outdatedOnGpu, emxArray_real_T *cpu_B,
              emxArray_real_T *gpu_B, boolean_T *B_outdatedOnGpu,
              emxArray_real_T *cpu_C, boolean_T *C_outdatedOnCpu,
              emxArray_real_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  nvtxRangePushA("#fcn#b_mtimes#" MW_AT_LOCATION);
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0) || (cpu_B->size[0] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_C, i, &d_emlrtRTEI);
    profileLoopStart("b_mtimes_loop_0", __LINE__, (cpu_A->size[0] - 1) + 1, "");
    for (i = 0; i < cpu_A->size[0]; i++) {
      cpu_C->data[i] = 0.0;
      *C_outdatedOnGpu = true;
    }
    profileLoopEnd();
  } else {
    real_T alpha1;
    real_T beta1;
    int32_T i;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_C, i, &c_emlrtRTEI);
    alpha1 = 1.0;
    beta1 = 0.0;
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_B, gpu_B, !*B_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    if (*B_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_B, cpu_B);
    }
    *B_outdatedOnGpu = false;
    nvtxMarkA("#cublasCheck#" MW_AT_LINE);
    cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            cpu_A->size[0], 1, cpu_A->size[1],
                            (double *)&alpha1, (double *)&gpu_A->data[0],
                            cpu_A->size[0], (double *)&gpu_B->data[0],
                            cpu_A->size[1], (double *)&beta1,
                            (double *)&gpu_C->data[0], cpu_A->size[0]),
                __FILE__, __LINE__);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
  nvtxRangePop();
}

//
//
void mtimes(emxArray_real_T *cpu_A, emxArray_real_T *gpu_A,
            boolean_T *A_outdatedOnGpu, emxArray_real_T *cpu_B,
            emxArray_real_T *gpu_B, boolean_T *B_outdatedOnGpu,
            emxArray_real_T *cpu_C, boolean_T *C_outdatedOnCpu,
            emxArray_real_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  nvtxRangePushA("#fcn#mtimes#" MW_AT_LOCATION);
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0) || (cpu_B->size[0] == 0) ||
      (cpu_B->size[1] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0] * cpu_C->size[1];
    cpu_C->size[0] = cpu_A->size[0];
    cpu_C->size[1] = cpu_B->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_C, i, &d_emlrtRTEI);
    profileLoopStart("mtimes_loop_0", __LINE__,
                     (cpu_A->size[0] * cpu_B->size[1] - 1) + 1, "");
    for (i = 0; i < cpu_A->size[0] * cpu_B->size[1]; i++) {
      cpu_C->data[i] = 0.0;
      *C_outdatedOnGpu = true;
    }
    profileLoopEnd();
  } else {
    real_T alpha1;
    real_T beta1;
    int32_T i;
    i = cpu_C->size[0] * cpu_C->size[1];
    cpu_C->size[0] = cpu_A->size[0];
    cpu_C->size[1] = cpu_B->size[1];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_C, i, &c_emlrtRTEI);
    alpha1 = 1.0;
    beta1 = 0.0;
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_B, gpu_B, !*B_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    if (*B_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_B, cpu_B);
    }
    *B_outdatedOnGpu = false;
    nvtxMarkA("#cublasCheck#" MW_AT_LINE);
    cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            cpu_A->size[0], cpu_B->size[1], cpu_A->size[1],
                            (double *)&alpha1, (double *)&gpu_A->data[0],
                            cpu_A->size[0], (double *)&gpu_B->data[0],
                            cpu_A->size[1], (double *)&beta1,
                            (double *)&gpu_C->data[0], cpu_A->size[0]),
                __FILE__, __LINE__);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
  nvtxRangePop();
}

} // namespace blas
} // namespace internal
} // namespace coder

// End of code generation (mtimes.cu)

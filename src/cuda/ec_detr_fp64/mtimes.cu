//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mtimes.cu
//
// Code generation for function 'mtimes'
//

// Include files
#include "mtimes.h"
#include "ec_detr_data.h"
#include "ec_detr_emxutil.h"
#include "ec_detr_mexutil.h"
#include "ec_detr_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"

// Variable Definitions
static emlrtRTEInfo
    d_emlrtRTEI{
        140,      // lineNo
        5,        // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

// Function Definitions
//
//
namespace coder {
namespace internal {
namespace blas {
void b_mtimes(emxArray_real_T *cpu_A, emxArray_real_T *gpu_A,
              boolean_T *A_outdatedOnGpu, emxArray_real_T *cpu_B,
              emxArray_real_T *gpu_B, boolean_T *B_outdatedOnGpu,
              emxArray_real_T *cpu_C, boolean_T *C_outdatedOnCpu,
              emxArray_real_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0) || (cpu_B->size[0] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    emxEnsureCapacity_real_T(cpu_C, i, &d_emlrtRTEI);
    for (i = 0; i < cpu_A->size[0]; i++) {
      cpu_C->data[i] = 0.0;
      *C_outdatedOnGpu = true;
    }
  } else {
    real_T alpha1;
    real_T beta1;
    int32_T i;
    i = cpu_C->size[0];
    cpu_C->size[0] = cpu_A->size[0];
    emxEnsureCapacity_real_T(cpu_C, i, &c_emlrtRTEI);
    alpha1 = 1.0;
    beta1 = 0.0;
    gpuEmxEnsureCapacity_real_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_B, gpu_B, !*B_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    if (*B_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_B, cpu_B);
    }
    *B_outdatedOnGpu = false;
    cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            cpu_A->size[0], 1, cpu_A->size[1],
                            (double *)&alpha1, (double *)&gpu_A->data[0],
                            cpu_A->size[0], (double *)&gpu_B->data[0],
                            cpu_A->size[1], (double *)&beta1,
                            (double *)&gpu_C->data[0], cpu_A->size[0]),
                __FILE__, __LINE__);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
}

//
//
void mtimes(emxArray_real_T *cpu_A, emxArray_real_T *gpu_A,
            boolean_T *A_outdatedOnGpu, emxArray_real_T *cpu_B,
            emxArray_real_T *gpu_B, boolean_T *B_outdatedOnGpu,
            emxArray_real_T *cpu_C, boolean_T *C_outdatedOnCpu,
            emxArray_real_T *gpu_C, boolean_T *C_outdatedOnGpu)
{
  if ((cpu_A->size[0] == 0) || (cpu_A->size[1] == 0) || (cpu_B->size[0] == 0) ||
      (cpu_B->size[1] == 0)) {
    int32_T i;
    *C_outdatedOnCpu = false;
    *C_outdatedOnGpu = false;
    i = cpu_C->size[0] * cpu_C->size[1];
    cpu_C->size[0] = cpu_A->size[0];
    cpu_C->size[1] = cpu_B->size[1];
    emxEnsureCapacity_real_T(cpu_C, i, &d_emlrtRTEI);
    for (i = 0; i < cpu_A->size[0] * cpu_B->size[1]; i++) {
      cpu_C->data[i] = 0.0;
      *C_outdatedOnGpu = true;
    }
  } else {
    real_T alpha1;
    real_T beta1;
    int32_T i;
    i = cpu_C->size[0] * cpu_C->size[1];
    cpu_C->size[0] = cpu_A->size[0];
    cpu_C->size[1] = cpu_B->size[1];
    emxEnsureCapacity_real_T(cpu_C, i, &c_emlrtRTEI);
    alpha1 = 1.0;
    beta1 = 0.0;
    gpuEmxEnsureCapacity_real_T(cpu_A, gpu_A, !*A_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_B, gpu_B, !*B_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_C, gpu_C, true);
    if (*A_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_A, cpu_A);
    }
    *A_outdatedOnGpu = false;
    if (*B_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_B, cpu_B);
    }
    *B_outdatedOnGpu = false;
    cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            cpu_A->size[0], cpu_B->size[1], cpu_A->size[1],
                            (double *)&alpha1, (double *)&gpu_A->data[0],
                            cpu_A->size[0], (double *)&gpu_B->data[0],
                            cpu_A->size[1], (double *)&beta1,
                            (double *)&gpu_C->data[0], cpu_A->size[0]),
                __FILE__, __LINE__);
    *C_outdatedOnGpu = false;
    *C_outdatedOnCpu = true;
  }
}

} // namespace blas
} // namespace internal
} // namespace coder

// End of code generation (mtimes.cu)

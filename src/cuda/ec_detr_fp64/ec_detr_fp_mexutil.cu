//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_detr_fp_mexutil.cu
//
// Code generation for function 'ec_detr_fp_mexutil'
//

// Include files
#include "ec_detr_fp_mexutil.h"
#include "ec_detr_fp_data.h"
#include "ec_detr_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cstdlib>
#include <cstring>

// Function Definitions
void b_raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                      const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  nvtxRangePushA("#fcn#b_raiseCudaError#" MW_AT_LOCATION);
  len = strlen(file);
  pn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  fn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

void checkCudaError(hipError_t errCode, const char_T *file, uint32_T b_line)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errCode != hipSuccess) {
    nvtxMarkA("#b_raiseCudaError#" MW_AT_LINE);
    b_raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                     hipGetErrorString(errCode));
  }
  nvtxRangePop();
}

int64_T computeEndIdx(int64_T start, int64_T end, int64_T stride)
{
  int64_T newEnd;
  nvtxRangePushA("#fcn#computeEndIdx#" MW_AT_LOCATION);
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  nvtxRangePop();
  return newEnd;
}

uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
  }
  numIters *= n;
  nvtxRangePop();
  return numIters;
}

uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  nvtxRangePop();
  return numIters;
}

void cublasCheck(hipblasStatus_t errCode, const char_T *file, uint32_T b_line)
{
  const char *errName;
  const char *errString;
  nvtxRangePushA("#fcn#cublasCheck#" MW_AT_LOCATION);
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    nvtxMarkA("#raiseCudaError#" MW_AT_LINE);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                  emxArray_int32_T *gpu, boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  int32_T *newData;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_int32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(int32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(int32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                 emxArray_real_T *gpu, boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxFree_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_int32_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                  const emxArray_int32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(int32_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                 const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyGpuToCpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxReset_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_int32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_int32_T));
  nvtxRangePop();
}

void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real_T));
  nvtxRangePop();
}

void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                    const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  nvtxRangePushA("#fcn#raiseCudaError#" MW_AT_LOCATION);
  len = strlen(file);
  pn = static_cast<char_T *>(std::calloc(static_cast<uint32_T>(len + 1UL), 1U));
  fn = static_cast<char_T *>(std::calloc(static_cast<uint32_T>(len + 1UL), 1U));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

// End of code generation (ec_detr_fp_mexutil.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_detr_fp_mexutil.cu
//
// Code generation for function 'ec_detr_fp_mexutil'
//

// Include files
#include "ec_detr_fp_mexutil.h"
#include "ec_detr_fp_data.h"
#include "ec_detr_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cstring>

// Function Definitions
void checkCudaError(hipError_t errorCode, const char_T *file, int32_T b_line)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errorCode != hipSuccess) {
    nvtxMarkA("#gpuThrowError#" MW_AT_LINE);
    gpuThrowError(errorCode, hipGetErrorName(errorCode),
                  hipGetErrorString(errorCode), file, b_line);
  }
  nvtxRangePop();
}

int64_T computeEndIdx(int64_T start, int64_T end, int64_T stride)
{
  int64_T newEnd;
  nvtxRangePushA("#fcn#computeEndIdx#" MW_AT_LOCATION);
  newEnd = -1L;
  if ((stride > 0L) && (start <= end)) {
    newEnd = (end - start) / stride;
  } else if ((stride < 0L) && (end <= start)) {
    newEnd = (start - end) / -stride;
  }
  nvtxRangePop();
  return newEnd;
}

uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  boolean_T overflow;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  overflow = false;
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
    overflow = (numIters > MAX_uint64_T / static_cast<uint64_T>(b_ub + 1));
  }
  numIters *= n;
  if (overflow) {
    nvtxMarkA("#gpuThrowError#" MW_AT_LINE);
    gpuThrowError(__FILE__, __LINE__);
  }
  nvtxRangePop();
  return numIters;
}

uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  nvtxRangePop();
  return numIters;
}

void gpuEmxEnsureCapacity_int32_T(const emxArray_int32_T *cpu,
                                  emxArray_int32_T *gpu, boolean_T needsCopy)
{
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  int32_T *newData;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_int32_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(int32_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(int32_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                 emxArray_real_T *gpu, boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (((totalSizeCpu == 0) && (cpu->allocatedSize > 0)) ||
      (gpu->allocatedSize < totalSizeCpu)) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && (totalSizeGpu > 0));
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxFree_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_int32_T#" MW_AT_LOCATION);
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real_T#" MW_AT_LOCATION);
  if (gpu->data && gpu->canFreeData && (gpu->data != (void *)4207599121UL)) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_int32_T(emxArray_int32_T *gpu,
                                  const emxArray_int32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_int32_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_int32_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (cpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(gpu->data, cpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(int32_T),
                   hipMemcpyHostToDevice),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                 const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (cpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(gpu->data, cpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(real_T),
                   hipMemcpyHostToDevice),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyGpuToCpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->data) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
    checkCudaError(
        hipMemcpy(cpu->data, gpu->data,
                   static_cast<uint32_T>(actualSize) * sizeof(real_T),
                   hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  nvtxRangePop();
}

void gpuEmxReset_int32_T(emxArray_int32_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_int32_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_int32_T));
  nvtxRangePop();
}

void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real_T));
  nvtxRangePop();
}

void gpuThrowError(const char_T *file, int32_T b_line)
{
  emlrtRTEInfo rtInfo;
  nvtxRangePushA("#fcn#gpuThrowError#" MW_AT_LOCATION);
  rtInfo.lineNo = b_line;
  rtInfo.colNo = 0;
  rtInfo.fName = "";
  rtInfo.pName = file;
  emlrtCUDAError(
      0U, (char_T *)"_",
      (char_T
           *)"Unable to launch kernel. Loop nest contains too many iterations.",
      &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

void gpuThrowError(uint32_T errorCode, const char_T *errorName,
                   const char_T *errorString, const char_T *file,
                   int32_T b_line)
{
  emlrtRTEInfo rtInfo;
  nvtxRangePushA("#fcn#gpuThrowError#" MW_AT_LOCATION);
  rtInfo.lineNo = b_line;
  rtInfo.colNo = 0;
  rtInfo.fName = "";
  rtInfo.pName = file;
  emlrtCUDAError(errorCode, (char_T *)errorName, (char_T *)errorString, &rtInfo,
                 emlrtRootTLSGlobal);
  nvtxRangePop();
}

// End of code generation (ec_detr_fp_mexutil.cu)

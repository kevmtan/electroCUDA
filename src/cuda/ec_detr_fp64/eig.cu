#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// eig.cu
//
// Code generation for function 'eig'
//

// Include files
#include "eig.h"
#include "ec_detr_fp_data.h"
#include "ec_detr_fp_emxutil.h"
#include "ec_detr_fp_mexutil.h"
#include "ec_detr_fp_types.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "lapacke.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>
#include <cstddef>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo e_emlrtRTEI{
    56,                                                            // lineNo
    24,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    60,                                                            // lineNo
    28,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    85,                                                            // lineNo
    9,                                                             // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    1,         // lineNo
    30,        // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    89,                                                            // lineNo
    13,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo
    j_emlrtRTEI{
        40,      // lineNo
        37,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo k_emlrtRTEI{
    47,        // lineNo
    20,        // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo
    l_emlrtRTEI{
        1,        // lineNo
        27,       // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo
    m_emlrtRTEI{
        99,      // lineNo
        24,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo n_emlrtRTEI{
    42,                                                              // lineNo
    9,                                                               // colNo
    "schur",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/schur.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    62,        // lineNo
    9,         // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo
    p_emlrtRTEI{
        102,     // lineNo
        21,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    q_emlrtRTEI{
        76,       // lineNo
        22,       // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo r_emlrtRTEI{
    28,                     // lineNo
    9,                      // colNo
    "eigHermitianStandard", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/private/"
    "eigHermitianStandard.m" // pName
};

static emlrtRTEInfo
    s_emlrtRTEI{
        131,     // lineNo
        29,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo t_emlrtRTEI{
    69,                                                              // lineNo
    13,                                                              // colNo
    "schur",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/schur.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    63,        // lineNo
    9,         // colNo
    "xsyheev", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
    "xsyheev.m" // pName
};

static emlrtRTEInfo
    v_emlrtRTEI{
        132,     // lineNo
        29,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo w_emlrtRTEI{
    46,                                                              // lineNo
    9,                                                               // colNo
    "schur",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/schur.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    33,                     // lineNo
    5,                      // colNo
    "eigHermitianStandard", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/private/"
    "eigHermitianStandard.m" // pName
};

static emlrtRTEInfo
    y_emlrtRTEI{
        86,       // lineNo
        9,        // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo
    ab_emlrtRTEI{
        134,     // lineNo
        35,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    bb_emlrtRTEI{
        87,       // lineNo
        9,        // colNo
        "xgehrd", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgehrd.m" // pName
    };

static emlrtRTEInfo cb_emlrtRTEI{
    23,                             // lineNo
    9,                              // colNo
    "eigRealSkewSymmetricStandard", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/private/"
    "eigRealSkewSymmetricStandard.m" // pName
};

static emlrtRTEInfo
    db_emlrtRTEI{
        111,      // lineNo
        29,       // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    eb_emlrtRTEI{
        112,      // lineNo
        29,       // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    fb_emlrtRTEI{
        160,     // lineNo
        13,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo gb_emlrtRTEI{
    24,                             // lineNo
    9,                              // colNo
    "eigRealSkewSymmetricStandard", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/private/"
    "eigRealSkewSymmetricStandard.m" // pName
};

static emlrtRTEInfo
    hb_emlrtRTEI{
        168,     // lineNo
        16,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    ib_emlrtRTEI{
        129,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    jb_emlrtRTEI{
        161,     // lineNo
        13,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo kb_emlrtRTEI{
    27,            // lineNo
    13,            // colNo
    "eigStandard", // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/private/"
    "eigStandard.m" // pName
};

static emlrtRTEInfo
    lb_emlrtRTEI{
        130,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo mb_emlrtRTEI{
    1,                                                             // lineNo
    20,                                                            // colNo
    "eig",                                                         // fName
    "/usr/local/MATLAB/R2024b/toolbox/eml/lib/matlab/matfun/eig.m" // pName
};

static emlrtRTEInfo
    nb_emlrtRTEI{
        99,      // lineNo
        1,       // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    ob_emlrtRTEI{
        131,     // lineNo
        5,       // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    pb_emlrtRTEI{
        132,     // lineNo
        5,       // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    qb_emlrtRTEI{
        134,     // lineNo
        10,      // colNo
        "xgeev", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xgeev.m" // pName
    };

static emlrtRTEInfo
    rb_emlrtRTEI{
        111,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

static emlrtRTEInfo
    sb_emlrtRTEI{
        112,      // lineNo
        9,        // colNo
        "xhseqr", // fName
        "/usr/local/MATLAB/R2024b/toolbox/eml/eml/+coder/+internal/+lapack/"
        "xhseqr.m" // pName
    };

// Function Definitions
//
//
namespace coder {
void eig(const emxArray_real_T *A, emxArray_creal_T *V, emxArray_creal_T *D)
{
  ptrdiff_t ihi_t;
  ptrdiff_t ilo_t;
  emxArray_creal_T *b_W;
  emxArray_real_T *U;
  emxArray_real_T *W;
  emxArray_real_T *b_A;
  emxArray_real_T *c_A;
  emxArray_real_T *d_A;
  emxArray_real_T *scale;
  emxArray_real_T *tau;
  emxArray_real_T *vright;
  emxArray_real_T *wi;
  emxArray_real_T *wimag;
  emxArray_real_T *wr;
  emxArray_real_T *wreal;
  real_T abnrm;
  real_T rconde;
  real_T rcondv;
  real_T vleft;
  int32_T j;
  int32_T n;
  nvtxRangePushA("#fcn#eig#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  n = A->size[0];
  j = V->size[0] * V->size[1];
  V->size[0] = A->size[0];
  V->size[1] = A->size[0];
  nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
  emxEnsureCapacity_creal_T(V, j, &e_emlrtRTEI);
  j = D->size[0] * D->size[1];
  D->size[0] = A->size[0];
  D->size[1] = A->size[0];
  nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
  emxEnsureCapacity_creal_T(D, j, &f_emlrtRTEI);
  if ((A->size[0] != 0) && (A->size[1] != 0)) {
    int32_T istart;
    int32_T nx;
    boolean_T p;
    nx = A->size[0] * A->size[1];
    p = true;
    profileLoopStart("eig_loop_0", __LINE__, (nx - 1) + 1, "");
    for (istart = 0; istart < nx; istart++) {
      if ((!p) ||
          (std::isinf(A->data[istart]) || std::isnan(A->data[istart]))) {
        p = false;
      }
    }
    profileLoopEnd();
    if (!p) {
      nx = A->size[0];
      istart = A->size[0];
      j = V->size[0] * V->size[1];
      V->size[0] = A->size[0];
      V->size[1] = A->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(V, j, &g_emlrtRTEI);
      profileLoopStart("eig_loop_1", __LINE__, (nx * istart - 1) + 1, "");
      for (j = 0; j < nx * istart; j++) {
        V->data[j].re = rtNaN;
        V->data[j].im = 0.0;
      }
      profileLoopEnd();
      nx = A->size[0];
      istart = A->size[0];
      j = D->size[0] * D->size[1];
      D->size[0] = A->size[0];
      D->size[1] = A->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(D, j, &i_emlrtRTEI);
      if (nx * istart - 1 >= 0) {
        std::memset(&D->data[0], 0,
                    static_cast<uint32_T>(nx * istart) * sizeof(creal_T));
      }
      profileLoopStart("eig_loop_5", __LINE__, (n - 1) + 1, "");
      for (istart = 0; istart < n; istart++) {
        D->data[istart + D->size[0] * istart].re = rtNaN;
        D->data[istart + D->size[0] * istart].im = 0.0;
      }
      profileLoopEnd();
    } else {
      int32_T exitg1;
      boolean_T exitg2;
      p = (A->size[0] == A->size[1]);
      if (p) {
        j = 0;
        exitg2 = false;
        nvtxRangePushA("#loop#eig_whileloop_4##" MW_AT_LINE);
        while ((!exitg2) && (j <= A->size[1] - 1)) {
          istart = 0;
          nvtxRangePushA("#loop#eig_whileloop_5##" MW_AT_LINE);
          do {
            exitg1 = 0;
            if (istart <= j) {
              if (!(A->data[istart + A->size[0] * j] ==
                    A->data[j + A->size[0] * istart])) {
                p = false;
                exitg1 = 1;
              } else {
                istart++;
              }
            } else {
              j++;
              exitg1 = 2;
            }
          } while (exitg1 == 0);
          nvtxRangePop();
          if (exitg1 == 1) {
            exitg2 = true;
          }
        }
        nvtxRangePop();
      }
      if (p) {
        ptrdiff_t info_t;
        n = A->size[0];
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&b_A, 2, &mb_emlrtRTEI, true);
        j = b_A->size[0] * b_A->size[1];
        b_A->size[0] = A->size[0];
        b_A->size[1] = A->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_A, j, &h_emlrtRTEI);
        profileLoopStart("eig_loop_3", __LINE__,
                         (A->size[0] * A->size[1] - 1) + 1, "");
        for (j = 0; j < A->size[0] * A->size[1]; j++) {
          b_A->data[j] = A->data[j];
        }
        ptrdiff_t n_t;
        profileLoopEnd();
        n_t = (ptrdiff_t)b_A->size[0];
        nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
        emxInit_real_T(&W, 1, &mb_emlrtRTEI, true);
        j = W->size[0];
        W->size[0] = b_A->size[0];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(W, j, &k_emlrtRTEI);
        info_t =
            LAPACKE_dsyev(102, 'V', 'L', n_t, &b_A->data[0], n_t, &W->data[0]);
        if ((int32_T)info_t < 0) {
          nx = W->size[0];
          j = W->size[0];
          W->size[0] = nx;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(W, j, &o_emlrtRTEI);
          profileLoopStart("eig_loop_8", __LINE__, (nx - 1) + 1, "");
          for (j = 0; j < nx; j++) {
            W->data[j] = rtNaN;
          }
          profileLoopEnd();
          nx = b_A->size[0];
          istart = b_A->size[1];
          j = b_A->size[0] * b_A->size[1];
          b_A->size[0] = nx;
          b_A->size[1] = istart;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_A, j, &u_emlrtRTEI);
          profileLoopStart("eig_loop_15", __LINE__, (nx * istart - 1) + 1, "");
          for (j = 0; j < nx * istart; j++) {
            b_A->data[j] = rtNaN;
          }
          profileLoopEnd();
        }
        j = D->size[0] * D->size[1];
        D->size[0] = A->size[0];
        D->size[1] = A->size[0];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(D, j, &r_emlrtRTEI);
        profileLoopStart("eig_loop_9", __LINE__,
                         (A->size[0] * A->size[0] - 1) + 1, "");
        for (j = 0; j < A->size[0] * A->size[0]; j++) {
          D->data[j].re = 0.0;
          D->data[j].im = 0.0;
        }
        profileLoopEnd();
        profileLoopStart("eig_loop_11", __LINE__, (n - 1) + 1, "");
        for (istart = 0; istart < n; istart++) {
          D->data[istart + D->size[0] * istart].re = W->data[istart];
          D->data[istart + D->size[0] * istart].im = 0.0;
        }
        profileLoopEnd();
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&W);
        j = V->size[0] * V->size[1];
        V->size[0] = b_A->size[0];
        V->size[1] = b_A->size[1];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(V, j, &x_emlrtRTEI);
        profileLoopStart("eig_loop_17", __LINE__,
                         (b_A->size[0] * b_A->size[1] - 1) + 1, "");
        for (j = 0; j < b_A->size[0] * b_A->size[1]; j++) {
          V->data[j].re = b_A->data[j];
          V->data[j].im = 0.0;
        }
        profileLoopEnd();
        nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
        emxFree_real_T(&b_A);
      } else {
        p = (A->size[0] == A->size[1]);
        if (p) {
          j = 0;
          exitg2 = false;
          nvtxRangePushA("#loop#eig_whileloop_2##" MW_AT_LINE);
          while ((!exitg2) && (j <= A->size[1] - 1)) {
            istart = 0;
            nvtxRangePushA("#loop#eig_whileloop_3##" MW_AT_LINE);
            do {
              exitg1 = 0;
              if (istart <= j) {
                if (!(A->data[istart + A->size[0] * j] ==
                      -A->data[j + A->size[0] * istart])) {
                  p = false;
                  exitg1 = 1;
                } else {
                  istart++;
                }
              } else {
                j++;
                exitg1 = 2;
              }
            } while (exitg1 == 0);
            nvtxRangePop();
            if (exitg1 == 1) {
              exitg2 = true;
            }
          }
          nvtxRangePop();
        }
        if (p) {
          nx = A->size[0] * A->size[1];
          profileLoopStart("eig_loop_2", __LINE__, (nx - 1) + 1, "");
          for (istart = 0; istart < nx; istart++) {
            if ((!p) ||
                (std::isinf(A->data[istart]) || std::isnan(A->data[istart]))) {
              p = false;
            }
          }
          profileLoopEnd();
          nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
          emxInit_real_T(&d_A, 2, &mb_emlrtRTEI, true);
          nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
          emxInit_real_T(&U, 2, &mb_emlrtRTEI, true);
          if (!p) {
            uint32_T dv_idx_0;
            uint32_T dv_idx_1;
            dv_idx_0 = static_cast<uint32_T>(A->size[0]);
            dv_idx_1 = static_cast<uint32_T>(A->size[1]);
            j = U->size[0] * U->size[1];
            U->size[0] = A->size[0];
            U->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(U, j, &n_emlrtRTEI);
            profileLoopStart("eig_loop_7", __LINE__,
                             (static_cast<int32_T>(dv_idx_0) *
                                  static_cast<int32_T>(dv_idx_1) -
                              1) +
                                 1,
                             "");
            for (j = 0; j < static_cast<int32_T>(dv_idx_0) *
                                static_cast<int32_T>(dv_idx_1);
                 j++) {
              U->data[j] = rtNaN;
            }
            profileLoopEnd();
            nx = A->size[0];
            if (A->size[0] > 1) {
              istart = 2;
              if (A->size[0] - 2 < A->size[1] - 1) {
                n = A->size[0] - 1;
              } else {
                n = A->size[1];
              }
              profileLoopStart("eig_loop_10", __LINE__, (n - 1) + 1, "");
              for (j = 0; j < n; j++) {
                int64_T b;
                nvtxMarkA("#computeEndIdx#" MW_AT_LINE);
                b = computeEndIdx(static_cast<int64_T>(istart),
                                  static_cast<int64_T>(nx), 1L);
                profileLoopStart("eig_loop_14", __LINE__, b + 1L, "");
                for (int64_T i{0L}; i <= b; i++) {
                  U->data[(static_cast<int32_T>(istart + i) + U->size[0] * j) -
                          1] = 0.0;
                }
                profileLoopEnd();
                istart++;
              }
              profileLoopEnd();
            }
            dv_idx_0 = static_cast<uint32_T>(A->size[0]);
            dv_idx_1 = static_cast<uint32_T>(A->size[1]);
            j = d_A->size[0] * d_A->size[1];
            d_A->size[0] = A->size[0];
            d_A->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(d_A, j, &w_emlrtRTEI);
            profileLoopStart("eig_loop_13", __LINE__,
                             (static_cast<int32_T>(dv_idx_0) *
                                  static_cast<int32_T>(dv_idx_1) -
                              1) +
                                 1,
                             "");
            for (j = 0; j < static_cast<int32_T>(dv_idx_0) *
                                static_cast<int32_T>(dv_idx_1);
                 j++) {
              d_A->data[j] = rtNaN;
            }
            profileLoopEnd();
          } else {
            ptrdiff_t b_n_t;
            ptrdiff_t e_info_t;
            j = d_A->size[0] * d_A->size[1];
            d_A->size[0] = A->size[0];
            d_A->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(d_A, j, &l_emlrtRTEI);
            profileLoopStart("eig_loop_6", __LINE__,
                             (A->size[0] * A->size[1] - 1) + 1, "");
            for (j = 0; j < A->size[0] * A->size[1]; j++) {
              d_A->data[j] = A->data[j];
            }
            profileLoopEnd();
            nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
            emxInit_real_T(&tau, 1, &mb_emlrtRTEI, true);
            j = tau->size[0];
            tau->size[0] = d_A->size[0] - 1;
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(tau, j, &q_emlrtRTEI);
            if (d_A->size[0] > 1) {
              ptrdiff_t b_info_t;
              b_info_t =
                  LAPACKE_dgehrd(102, (ptrdiff_t)d_A->size[0], (ptrdiff_t)1,
                                 (ptrdiff_t)d_A->size[0], &d_A->data[0],
                                 (ptrdiff_t)d_A->size[0], &tau->data[0]);
              if ((int32_T)b_info_t != 0) {
                nx = d_A->size[0];
                istart = d_A->size[1];
                j = d_A->size[0] * d_A->size[1];
                d_A->size[0] = nx;
                d_A->size[1] = istart;
                nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
                emxEnsureCapacity_real_T(d_A, j, &y_emlrtRTEI);
                profileLoopStart("eig_loop_16", __LINE__, (nx * istart - 1) + 1,
                                 "");
                for (j = 0; j < nx * istart; j++) {
                  d_A->data[j] = rtNaN;
                }
                profileLoopEnd();
                nx = tau->size[0];
                j = tau->size[0];
                tau->size[0] = nx;
                nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
                emxEnsureCapacity_real_T(tau, j, &bb_emlrtRTEI);
                profileLoopStart("eig_loop_19", __LINE__, (nx - 1) + 1, "");
                for (j = 0; j < nx; j++) {
                  tau->data[j] = rtNaN;
                }
                profileLoopEnd();
              }
            }
            j = U->size[0] * U->size[1];
            U->size[0] = d_A->size[0];
            U->size[1] = d_A->size[1];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(U, j, &t_emlrtRTEI);
            profileLoopStart("eig_loop_12", __LINE__,
                             (d_A->size[0] * d_A->size[1] - 1) + 1, "");
            for (j = 0; j < d_A->size[0] * d_A->size[1]; j++) {
              U->data[j] = d_A->data[j];
            }
            profileLoopEnd();
            if (A->size[0] == 1) {
              U->data[0] = 1.0;
            } else {
              ptrdiff_t c_info_t;
              c_info_t =
                  LAPACKE_dorghr(102, (ptrdiff_t)A->size[0], (ptrdiff_t)1,
                                 (ptrdiff_t)A->size[0], &U->data[0],
                                 (ptrdiff_t)A->size[0], &tau->data[0]);
              if ((int32_T)c_info_t != 0) {
                nx = U->size[0];
                istart = U->size[1];
                j = U->size[0] * U->size[1];
                U->size[0] = nx;
                U->size[1] = istart;
                nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
                emxEnsureCapacity_real_T(U, j, &t_emlrtRTEI);
                profileLoopStart("eig_loop_20", __LINE__, (nx * istart - 1) + 1,
                                 "");
                for (j = 0; j < nx * istart; j++) {
                  U->data[j] = rtNaN;
                }
                profileLoopEnd();
              }
            }
            nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
            emxFree_real_T(&tau);
            b_n_t = (ptrdiff_t)d_A->size[0];
            nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
            emxInit_real_T(&wr, 2, &rb_emlrtRTEI, true);
            j = wr->size[0] * wr->size[1];
            wr->size[0] = 1;
            wr->size[1] = d_A->size[0];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(wr, j, &db_emlrtRTEI);
            nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
            emxInit_real_T(&wi, 2, &sb_emlrtRTEI, true);
            j = wi->size[0] * wi->size[1];
            wi->size[0] = 1;
            wi->size[1] = d_A->size[0];
            nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
            emxEnsureCapacity_real_T(wi, j, &eb_emlrtRTEI);
            if (d_A->size[0] <= 1) {
              n = 1;
            } else {
              n = d_A->size[0];
            }
            e_info_t = LAPACKE_dhseqr(102, 'S', 'V', b_n_t, (ptrdiff_t)1,
                                      (ptrdiff_t)d_A->size[0], &d_A->data[0],
                                      b_n_t, &wr->data[0], &wi->data[0],
                                      &U->data[0], (ptrdiff_t)n);
            nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
            emxFree_real_T(&wi);
            nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
            emxFree_real_T(&wr);
            if ((int32_T)e_info_t < 0) {
              nx = d_A->size[0];
              istart = d_A->size[1];
              j = d_A->size[0] * d_A->size[1];
              d_A->size[0] = nx;
              d_A->size[1] = istart;
              nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
              emxEnsureCapacity_real_T(d_A, j, &ib_emlrtRTEI);
              profileLoopStart("eig_loop_29", __LINE__, (nx * istart - 1) + 1,
                               "");
              for (j = 0; j < nx * istart; j++) {
                d_A->data[j] = rtNaN;
              }
              profileLoopEnd();
              nx = U->size[0];
              istart = U->size[1];
              j = U->size[0] * U->size[1];
              U->size[0] = nx;
              U->size[1] = istart;
              nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
              emxEnsureCapacity_real_T(U, j, &lb_emlrtRTEI);
              profileLoopStart("eig_loop_32", __LINE__, (nx * istart - 1) + 1,
                               "");
              for (j = 0; j < nx * istart; j++) {
                U->data[j] = rtNaN;
              }
              profileLoopEnd();
            }
          }
          n = d_A->size[0];
          j = D->size[0] * D->size[1];
          D->size[0] = d_A->size[0];
          D->size[1] = d_A->size[0];
          nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
          emxEnsureCapacity_creal_T(D, j, &cb_emlrtRTEI);
          profileLoopStart("eig_loop_18", __LINE__,
                           (d_A->size[0] * d_A->size[0] - 1) + 1, "");
          for (j = 0; j < d_A->size[0] * d_A->size[0]; j++) {
            D->data[j].re = 0.0;
            D->data[j].im = 0.0;
          }
          profileLoopEnd();
          istart = 1;
          nvtxRangePushA("#loop#eig_whileloop_1##" MW_AT_LINE);
          while (istart <= n) {
            if ((istart != n) &&
                (d_A->data[istart + d_A->size[0] * (istart - 1)] != 0.0)) {
              vleft = std::abs(d_A->data[istart + d_A->size[0] * (istart - 1)]);
              D->data[(istart + D->size[0] * (istart - 1)) - 1].re = 0.0;
              D->data[(istart + D->size[0] * (istart - 1)) - 1].im = vleft;
              D->data[istart + D->size[0] * istart].re = 0.0;
              D->data[istart + D->size[0] * istart].im = -vleft;
              istart += 2;
            } else {
              istart++;
            }
          }
          nvtxRangePop();
          j = V->size[0] * V->size[1];
          V->size[0] = U->size[0];
          V->size[1] = U->size[1];
          nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
          emxEnsureCapacity_creal_T(V, j, &gb_emlrtRTEI);
          profileLoopStart("eig_loop_22", __LINE__,
                           (U->size[0] * U->size[1] - 1) + 1, "");
          for (j = 0; j < U->size[0] * U->size[1]; j++) {
            V->data[j].re = U->data[j];
            V->data[j].im = 0.0;
          }
          profileLoopEnd();
          nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
          emxFree_real_T(&U);
          j = 1;
          n = d_A->size[0];
          nvtxRangePushA("#loop#eig_whileloop_0##" MW_AT_LINE);
          while (j <= n) {
            if ((j != n) && (d_A->data[j + d_A->size[0] * (j - 1)] != 0.0)) {
              if (d_A->data[j + d_A->size[0] * (j - 1)] < 0.0) {
                nx = 1;
              } else {
                nx = -1;
              }
              profileLoopStart("eig_loop_28", __LINE__, (n - 1) + 1, "");
              for (istart = 0; istart < n; istart++) {
                vleft = V->data[istart + V->size[0] * (j - 1)].re;
                abnrm = static_cast<real_T>(nx) *
                        V->data[istart + V->size[0] * j].re;
                if (abnrm == 0.0) {
                  V->data[istart + V->size[0] * (j - 1)].re =
                      vleft / 1.4142135623730951;
                  V->data[istart + V->size[0] * (j - 1)].im = 0.0;
                } else if (vleft == 0.0) {
                  V->data[istart + V->size[0] * (j - 1)].re = 0.0;
                  V->data[istart + V->size[0] * (j - 1)].im =
                      abnrm / 1.4142135623730951;
                } else {
                  V->data[istart + V->size[0] * (j - 1)].re =
                      vleft / 1.4142135623730951;
                  V->data[istart + V->size[0] * (j - 1)].im =
                      abnrm / 1.4142135623730951;
                }
                V->data[istart + V->size[0] * j].re =
                    V->data[istart + V->size[0] * (j - 1)].re;
                V->data[istart + V->size[0] * j].im =
                    -V->data[istart + V->size[0] * (j - 1)].im;
              }
              profileLoopEnd();
              j += 2;
            } else {
              j++;
            }
          }
          nvtxRangePop();
          nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
          emxFree_real_T(&d_A);
        } else {
          ptrdiff_t d_info_t;
          n = A->size[0];
          nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
          emxInit_real_T(&c_A, 2, &j_emlrtRTEI, true);
          j = c_A->size[0] * c_A->size[1];
          c_A->size[0] = A->size[0];
          c_A->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(c_A, j, &j_emlrtRTEI);
          profileLoopStart("eig_loop_4", __LINE__,
                           (A->size[0] * A->size[1] - 1) + 1, "");
          for (j = 0; j < A->size[0] * A->size[1]; j++) {
            c_A->data[j] = A->data[j];
          }
          profileLoopEnd();
          nx = A->size[1] - 1;
          nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
          emxInit_real_T(&scale, 1, &nb_emlrtRTEI, true);
          j = scale->size[0];
          scale->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(scale, j, &m_emlrtRTEI);
          nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
          emxInit_creal_T(&b_W, 1, &mb_emlrtRTEI, true);
          j = b_W->size[0];
          b_W->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
          emxEnsureCapacity_creal_T(b_W, j, &p_emlrtRTEI);
          nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
          emxInit_real_T(&wreal, 1, &ob_emlrtRTEI, true);
          j = wreal->size[0];
          wreal->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(wreal, j, &s_emlrtRTEI);
          nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
          emxInit_real_T(&wimag, 1, &pb_emlrtRTEI, true);
          j = wimag->size[0];
          wimag->size[0] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(wimag, j, &v_emlrtRTEI);
          nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
          emxInit_real_T(&vright, 2, &qb_emlrtRTEI, true);
          j = vright->size[0] * vright->size[1];
          vright->size[0] = A->size[1];
          vright->size[1] = A->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(vright, j, &ab_emlrtRTEI);
          d_info_t = LAPACKE_dgeevx(
              102, 'B', 'N', 'V', 'N', (ptrdiff_t)A->size[1], &c_A->data[0],
              (ptrdiff_t)A->size[0], &wreal->data[0], &wimag->data[0], &vleft,
              (ptrdiff_t)1, &vright->data[0], (ptrdiff_t)A->size[1], &ilo_t,
              &ihi_t, &scale->data[0], &abnrm, &rconde, &rcondv);
          nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
          emxFree_real_T(&scale);
          nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
          emxFree_real_T(&c_A);
          if ((int32_T)d_info_t < 0) {
            j = b_W->size[0];
            b_W->size[0] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
            emxEnsureCapacity_creal_T(b_W, j, &fb_emlrtRTEI);
            profileLoopStart("eig_loop_23", __LINE__, (A->size[1] - 1) + 1, "");
            for (j = 0; j < A->size[1]; j++) {
              b_W->data[j].re = rtNaN;
              b_W->data[j].im = 0.0;
            }
            profileLoopEnd();
            nx = A->size[1];
            istart = A->size[1];
            j = V->size[0] * V->size[1];
            V->size[0] = A->size[1];
            V->size[1] = A->size[1];
            nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
            emxEnsureCapacity_creal_T(V, j, &jb_emlrtRTEI);
            profileLoopStart("eig_loop_26", __LINE__, (nx * istart - 1) + 1,
                             "");
            for (j = 0; j < nx * istart; j++) {
              V->data[j].re = rtNaN;
              V->data[j].im = 0.0;
            }
            profileLoopEnd();
          } else {
            profileLoopStart("eig_loop_21", __LINE__, nx + 1, "");
            for (istart = 0; istart <= nx; istart++) {
              b_W->data[istart].re = wreal->data[istart];
              b_W->data[istart].im = wimag->data[istart];
            }
            profileLoopEnd();
            j = V->size[0] * V->size[1];
            V->size[0] = vright->size[0];
            V->size[1] = vright->size[1];
            nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
            emxEnsureCapacity_creal_T(V, j, &hb_emlrtRTEI);
            profileLoopStart("eig_loop_24", __LINE__,
                             (vright->size[0] * vright->size[1] - 1) + 1, "");
            for (j = 0; j < vright->size[0] * vright->size[1]; j++) {
              V->data[j].re = vright->data[j];
              V->data[j].im = 0.0;
            }
            profileLoopEnd();
            profileLoopStart("eig_loop_25", __LINE__, (nx - 1) + 1, "");
            for (istart = 0; istart < nx; istart++) {
              if ((wimag->data[istart] > 0.0) &&
                  (wimag->data[istart + 1] < 0.0)) {
                profileLoopStart("eig_loop_27", __LINE__, nx + 1, "");
                for (j = 0; j <= nx; j++) {
                  vleft = V->data[j + V->size[0] * istart].re;
                  abnrm = V->data[j + V->size[0] * (istart + 1)].re;
                  V->data[j + V->size[0] * istart].im = abnrm;
                  V->data[j + V->size[0] * (istart + 1)].re = vleft;
                  V->data[j + V->size[0] * (istart + 1)].im = -abnrm;
                }
                profileLoopEnd();
              }
            }
            profileLoopEnd();
          }
          nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
          emxFree_real_T(&vright);
          nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
          emxFree_real_T(&wimag);
          nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
          emxFree_real_T(&wreal);
          j = D->size[0] * D->size[1];
          D->size[0] = A->size[0];
          D->size[1] = A->size[0];
          nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
          emxEnsureCapacity_creal_T(D, j, &kb_emlrtRTEI);
          profileLoopStart("eig_loop_30", __LINE__,
                           (A->size[0] * A->size[0] - 1) + 1, "");
          for (j = 0; j < A->size[0] * A->size[0]; j++) {
            D->data[j].re = 0.0;
            D->data[j].im = 0.0;
          }
          profileLoopEnd();
          profileLoopStart("eig_loop_31", __LINE__, (n - 1) + 1, "");
          for (istart = 0; istart < n; istart++) {
            D->data[istart + D->size[0] * istart] = b_W->data[istart];
          }
          profileLoopEnd();
          nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
          emxFree_creal_T(&b_W);
        }
      }
    }
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (eig.cu)

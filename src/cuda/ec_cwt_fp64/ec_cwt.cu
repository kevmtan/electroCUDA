#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_cwt.cu
//
// Code generation for function 'ec_cwt'
//

// Include files
#include "ec_cwt.h"
#include "ResampleParser.h"
#include "conv2.h"
#include "cwtfilterbank.h"
#include "ec_cwt_data.h"
#include "ec_cwt_emxutil.h"
#include "ec_cwt_mexutil.h"
#include "ec_cwt_types.h"
#include "rt_nonfinite.h"
#include "uniformResampleKernel.h"
#include "wt.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hip/hip_math_constants.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    25,                                                 // lineNo
    1,                                                  // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    1,                                                  // lineNo
    22,                                                 // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    37,                                                 // lineNo
    21,                                                 // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    37,                                                 // lineNo
    5,                                                  // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    43,                                                 // lineNo
    9,                                                  // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    154,                                                        // lineNo
    1,                                                          // colNo
    "resample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/resample.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    38,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    21,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    41,                                                 // lineNo
    9,                                                  // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    31,                      // lineNo
    29,                      // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    41,                      // lineNo
    5,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    22,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    23,                                                 // lineNo
    1,                                                  // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    37,                                                 // lineNo
    11,                                                 // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    37,                                                 // lineNo
    15,                                                 // colNo
    "ec_cwt",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_cwt.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    15,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

// Function Declarations
static __global__ void ec_cwt_kernel1(const int32_T ch, const emxArray_real_T x,
                                      const int32_T b_x, emxArray_real_T c_x,
                                      int32_T x_dim0);

static __global__ void ec_cwt_kernel10(const emxArray_real_T xCh,
                                       const int32_T b_xCh, const int32_T c_xCh,
                                       emxArray_real_T opts_x,
                                       int32_T opts_x_dim0, int32_T xCh_dim0);

static __global__ void ec_cwt_kernel2(const emxArray_creal_T x, const int32_T b,
                                      emxArray_real_T y);

static __global__ void ec_cwt_kernel3(const emxArray_real_T y,
                                      const int32_T b_y, const int32_T c_y,
                                      emxArray_real_T xCh, int32_T xCh_dim0,
                                      int32_T y_dim0);

static __global__ void ec_cwt_kernel4(const emxArray_real_T xCh,
                                      const int32_T b, emxArray_real_T opts_x);

static __global__ void ec_cwt_kernel5(const emxArray_real_T w, const int32_T nx,
                                      const int32_T b, emxArray_real_T y);

static __global__ void ec_cwt_kernel6(emxArray_real_T y);

static __global__ void ec_cwt_kernel7(const emxArray_real_T y,
                                      const emxArray_real_T b_y,
                                      const real_T nlast, const int32_T ii,
                                      const int32_T b, emxArray_real_T b_y1,
                                      int32_T y1_dim0);

static __global__ void ec_cwt_kernel8(const int32_T b, emxArray_real_T y);

static __global__ void ec_cwt_kernel9(const emxArray_real_T xCh,
                                      const int32_T b_xCh,
                                      emxArray_real_T opts_x);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel1(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T c_x, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_x.data[i1] = x.data[i1 + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel10(
    const emxArray_real_T xCh, const int32_T b_xCh, const int32_T c_xCh,
    emxArray_real_T opts_x, int32_T opts_x_dim0, int32_T xCh_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_xCh) + 1UL) *
                (static_cast<uint64_T>(b_xCh) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_xCh) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_xCh) + 1UL));
    opts_x.data[k + opts_x_dim0 * i1] = xCh.data[i1 + xCh_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel2(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel3(
    const emxArray_real_T y, const int32_T b_y, const int32_T c_y,
    emxArray_real_T xCh, int32_T xCh_dim0, int32_T y_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c_y) + 1UL) * (static_cast<uint64_T>(b_y) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_y) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_y) + 1UL));
    xCh.data[k + xCh_dim0 * i1] = y.data[i1 + y_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel4(
    const emxArray_real_T xCh, const int32_T b, emxArray_real_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = xCh.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel5(
    const emxArray_real_T w, const int32_T nx, const int32_T b,
    emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = w.data[nx * i1];
  }
}

static __global__ __launch_bounds__(32,
                                    1) void ec_cwt_kernel6(emxArray_real_T y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    y.data[0] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel7(
    const emxArray_real_T y, const emxArray_real_T b_y, const real_T nlast,
    const int32_T ii, const int32_T b, emxArray_real_T b_y1, int32_T y1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1 + y1_dim0 * ii] =
        y.data[static_cast<int32_T>(nlast + b_y.data[i1]) - 1];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void ec_cwt_kernel8(const int32_T b,
                                                           emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<real_T>(i1) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_cwt_kernel9(
    const emxArray_real_T xCh, const int32_T b_xCh, emxArray_real_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_xCh);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = xCh.data[i1];
  }
}

//
// function [y,freqs] = ec_cwt(x,fs,fLims,fVoices,ds)
void ec_cwt(const emxArray_real_T *cpu_x, real_T fs, const real_T fLims[2],
            real_T fVoices, const real_T ds[2], emxArray_cell_wrap_0 *y,
            emxArray_real_T *freqs)
{
  static const int32_T iv1[2]{1, 7};
  static const int32_T iv2[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  coder::b_signal::internal::resample::ResampleParser expl_temp;
  coder::cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal_T c_gpu_x;
  emxArray_creal_T *b_cpu_x;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_y;
  emxArray_real_T b_yCol;
  emxArray_real_T c_gpu_y;
  emxArray_real_T gpu_opts_x;
  emxArray_real_T gpu_w;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xCh;
  emxArray_real_T gpu_y;
  emxArray_real_T gpu_y1;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_x;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_opts_x;
  emxArray_real_T *cpu_w;
  emxArray_real_T *cpu_xCh;
  emxArray_real_T *cpu_y;
  emxArray_real_T *cpu_y1;
  emxArray_real_T *yCol;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  real_T Ly;
  int32_T b_expl_temp[1];
  int32_T exponent;
  int32_T i;
  int32_T i1;
  int32_T i3;
  int32_T i4;
  char_T unusedExpr[23];
  boolean_T doDownsample;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  gpuEmxReset_real_T(&b_gpu_x);
  gpuEmxReset_real_T(&c_gpu_y);
  gpuEmxReset_real_T(&b_gpu_y);
  gpuEmxReset_real_T(&gpu_w);
  gpuEmxReset_real_T(&gpu_y1);
  gpuEmxReset_real_T(&gpu_opts_x);
  gpuEmxReset_real_T(&gpu_xCh);
  gpuEmxReset_creal_T(&c_gpu_x);
  gpuEmxReset_real_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_x);
  w_outdatedOnCpu = false;
  w_outdatedOnGpu = false;
  x_outdatedOnGpu = true;
  y_needsGpuEnsureCapacity = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInitStruct_cwtfilterbank(&fb, &r_emlrtRTEI, true);
  //  Continuous wavelet transform - CUDA binary wrapper (double-precision,
  //  FP64) CWT uses morse wavelets, as they account for unequal
  //  variance-covariance across freqs. L1-norm is applied to mitigate 1/f decay
  //  of neuronal field potentials. Log-transform & robust z-scoring converts
  //  absolute valued CWT outputs into normal distribution.
  //
  //  Kevin Tan, 2022 (kevmtan.github.io)
  //  Input validation
  // 'ec_cwt:11' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_cwt:12' fs (1,1) double = 1000
  //  Sampling rate
  // 'ec_cwt:13' fLims (1,2) double = [1 300]
  //  Frequency limits
  // 'ec_cwt:14' fVoices (1,1) double = 10
  //  Voices per octave
  // 'ec_cwt:15' ds (1,2) double = [1 1]
  //  Frequency limits
  // 'ec_cwt:17' coder.gpu.kernelfun;
  // 'ec_cwt:18' nFrames = height(x);
  // 'ec_cwt:19' nChs = width(x);
  // 'ec_cwt:20' doDownsample = ds(2)>ds(1);
  doDownsample = (ds[1] > ds[0]);
  //  Prep CWT
  // 'ec_cwt:23' fb = cwtfilterbank(Wavelet="Morse",SignalLength=nFrames,...
  // 'ec_cwt:24'
  // SamplingFrequency=fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
  coder::cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(cpu_x->size[0]),
                                     fs, fLims, fVoices);
  // 'ec_cwt:25' freqs = centerFrequencies(fb);
  i = freqs->size[0];
  freqs->size[0] = fb.WaveletCenterFrequencies->size[0];
  emxEnsureCapacity_real_T(freqs, i, &emlrtRTEI);
  for (i = 0; i < fb.WaveletCenterFrequencies->size[0]; i++) {
    freqs->data[i] = fb.WaveletCenterFrequencies->data[i];
  }
  //  nFreqs = uint32(numel(freqs));
  //  nFrames = uint32(numel(1:ds:nFrames));
  //  Preallocate
  // 'ec_cwt:30' tmp = cell(1,nChs);
  i = y->size[0] * y->size[1];
  y->size[0] = 1;
  y->size[1] = cpu_x->size[1];
  emxEnsureCapacity_cell_wrap_0(y, i, &b_emlrtRTEI);
  // 'ec_cwt:31' y = coder.nullcopy(tmp);
  //  y = coder.nullcopy(zeros([nFrames nChs nFreqs],like=x)); % Preallocate
  //  output Channel loop
  // 'ec_cwt:35' for ch = 1:nChs
  i1 = cpu_x->size[1];
  emxInit_real_T(&cpu_y, 2, &s_emlrtRTEI, true);
  emxInit_creal_T(&b_cpu_x, 2, &t_emlrtRTEI, true);
  emxInit_real_T(&cpu_xCh, 2, &e_emlrtRTEI, true);
  emxInit_real_T(&cpu_opts_x, 2, &g_emlrtRTEI, true);
  emxInit_real_T(&cpu_y1, 2, &u_emlrtRTEI, true);
  emxInit_real_T(&yCol, 1, &v_emlrtRTEI, true);
  emxInit_real_T(&cpu_w, 1, &o_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_y, 1, &o_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_y, 2, &w_emlrtRTEI, true);
  emxInitStruct_ResampleParser(&expl_temp, &g_emlrtRTEI, true);
  emxInit_real_T(&c_cpu_x, 1, &c_emlrtRTEI, true);
  for (int32_T ch{0}; ch < i1; ch++) {
    int32_T f_y;
    int32_T nx;
    uint32_T b_dv[2];
    boolean_T validLaunchParams;
    boolean_T xCh_outdatedOnCpu;
    //  CWT
    // 'ec_cwt:37' xCh = abs(wt(fb,x(:,ch)))';
    i = c_cpu_x->size[0];
    c_cpu_x->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real_T(c_cpu_x, i, &c_emlrtRTEI);
    i = cpu_x->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    if (x_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real_T(c_cpu_x, &b_gpu_x, true);
    if (x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwt_kernel1<<<grid, block>>>(ch, gpu_x, i, b_gpu_x, cpu_x->size[0U]);
    }
    gpuEmxMemcpyGpuToCpu_real_T(c_cpu_x, &b_gpu_x);
    coder::cwtfilterbank_wt(&fb, c_cpu_x, b_cpu_x);
    nx = b_cpu_x->size[0] * b_cpu_x->size[1];
    for (i = 0; i < 2; i++) {
      b_dv[i] = static_cast<uint32_T>(b_cpu_x->size[i]);
    }
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = static_cast<int32_T>(b_dv[0]);
    cpu_y->size[1] = static_cast<int32_T>(b_dv[1]);
    emxEnsureCapacity_real_T(cpu_y, i, &d_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_creal_T(b_cpu_x, &c_gpu_x, false);
    gpuEmxEnsureCapacity_real_T(cpu_y, &gpu_y, true);
    gpuEmxMemcpyCpuToGpu_creal_T(&c_gpu_x, b_cpu_x);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwt_kernel2<<<grid, block>>>(c_gpu_x, nx - 1, gpu_y);
    }
    i = cpu_xCh->size[0] * cpu_xCh->size[1];
    cpu_xCh->size[0] = cpu_y->size[1];
    cpu_xCh->size[1] = cpu_y->size[0];
    emxEnsureCapacity_real_T(cpu_xCh, i, &e_emlrtRTEI);
    i = cpu_y->size[0] - 1;
    f_y = cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i, f_y), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_xCh, &gpu_xCh, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_cwt_kernel3<<<grid, block>>>(gpu_y, f_y, i, gpu_xCh, cpu_xCh->size[0U],
                                      cpu_y->size[0U]);
    }
    xCh_outdatedOnCpu = true;
    //  CWT
    //  Downsample
    // 'ec_cwt:40' if doDownsample
    if (doDownsample) {
      real_T xCh[2];
      real_T absx;
      real_T dlast;
      real_T n;
      real_T x;
      real_T xin;
      int32_T opts_dim;
      // 'ec_cwt:41' y{ch} = resample(xCh,ds(1),ds(2));
      if ((cpu_xCh->size[0] == 1) || (cpu_xCh->size[1] == 1)) {
        opts_dim = 1;
      } else {
        opts_dim = 2;
        if (cpu_xCh->size[0] != 1) {
          opts_dim = 1;
        }
      }
      xCh_outdatedOnCpu = (cpu_xCh->size[0] == 1);
      if (opts_dim == 1) {
        if ((cpu_xCh->size[0] == 1) || (cpu_xCh->size[1] == 1)) {
          if ((cpu_xCh->size[0] == 0) || (cpu_xCh->size[1] == 0)) {
            f_y = 0;
          } else {
            nx = cpu_xCh->size[0];
            f_y = cpu_xCh->size[1];
            if (nx >= f_y) {
              f_y = nx;
            }
          }
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
          cpu_opts_x->size[0] = f_y;
          cpu_opts_x->size[1] = 1;
          emxEnsureCapacity_real_T(cpu_opts_x, i, &g_emlrtRTEI);
          mwGetLaunchParameters1D(computeNumIters(f_y - 1), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real_T(cpu_opts_x, &gpu_opts_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_cwt_kernel4<<<grid, block>>>(gpu_xCh, f_y - 1, gpu_opts_x);
          }
        } else {
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
          cpu_opts_x->size[0] = cpu_xCh->size[0];
          cpu_opts_x->size[1] = cpu_xCh->size[1];
          emxEnsureCapacity_real_T(cpu_opts_x, i, &g_emlrtRTEI);
          i = cpu_xCh->size[0] * cpu_xCh->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real_T(cpu_opts_x, &gpu_opts_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_cwt_kernel9<<<grid, block>>>(gpu_xCh, i, gpu_opts_x);
          }
        }
      } else {
        i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
        cpu_opts_x->size[0] = cpu_xCh->size[1];
        cpu_opts_x->size[1] = cpu_xCh->size[0];
        emxEnsureCapacity_real_T(cpu_opts_x, i, &g_emlrtRTEI);
        i = cpu_xCh->size[0] - 1;
        f_y = cpu_xCh->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(i, f_y), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real_T(cpu_opts_x, &gpu_opts_x, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_cwt_kernel10<<<grid, block>>>(gpu_xCh, f_y, i, gpu_opts_x,
                                           cpu_opts_x->size[0U],
                                           cpu_xCh->size[0U]);
        }
      }
      x = ds[0] / ds[1];
      xin = x;
      absx = std::abs(x);
      if (std::isinf(absx) || std::isnan(absx)) {
        absx = rtNaN;
      } else if (absx < 4.4501477170144028E-308) {
        absx = 4.94065645841247E-324;
      } else {
        std::frexp(absx, &exponent);
        absx = std::ldexp(1.0, exponent - 53);
      }
      absx = std::fmax(1.0E-12, absx);
      if (std::isinf(x) || std::isnan(x)) {
        if (!std::isnan(x)) {
          if (x < 0.0) {
            n = -1.0;
          } else {
            n = 1.0;
          }
        } else {
          n = 0.0;
        }
        absx = 0.0;
      } else {
        real_T d;
        real_T nlast;
        n = 1.0;
        d = 0.0;
        nlast = 0.0;
        dlast = 1.0;
        real_T rndx;
        int32_T exitg1;
        do {
          exitg1 = 0;
          rndx = std::round(x);
          if (!std::isinf(x)) {
            real_T tmp;
            x -= rndx;
            tmp = nlast;
            nlast = n;
            n = n * rndx + tmp;
            tmp = dlast;
            dlast = d;
            d = d * rndx + tmp;
          } else {
            nlast = n;
            dlast = d;
            n = x;
            d = 0.0;
          }
          if ((x == 0.0) || (std::abs(n / d - xin) <= absx)) {
            exitg1 = 1;
          } else {
            x = 1.0 / x;
          }
        } while (exitg1 == 0);
        if (std::isnan(d)) {
          x = rtNaN;
        } else if (d < 0.0) {
          x = -1.0;
        } else {
          x = (d > 0.0);
        }
        n /= x;
        absx = std::abs(d);
      }
      for (i = 0; i < 2; i++) {
        xCh[i] = cpu_xCh->size[i];
      }
      gpuEmxMemcpyGpuToCpu_real_T(cpu_opts_x, &gpu_opts_x);
      coder::b_signal::internal::resample::c_ResampleParser_createFilterAn(
          opts_dim, n, absx, xCh, xCh_outdatedOnCpu, cpu_opts_x, &expl_temp);
      absx = expl_temp.p;
      n = expl_temp.q;
      if (expl_temp.dim == 1) {
        if (expl_temp.isRowVectorInput) {
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = expl_temp.x->size[0];
          cpu_y1->size[1] = expl_temp.x->size[1];
          emxEnsureCapacity_real_T(cpu_y1, i, &i_emlrtRTEI);
          for (i = 0; i < expl_temp.x->size[0] * expl_temp.x->size[1]; i++) {
            cpu_y1->data[i] = expl_temp.x->data[i];
          }
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_y1, expl_temp.p, expl_temp.q, expl_temp.filterWithPadding,
              expl_temp.filterDelay);
          if ((cpu_y1->size[0] == 0) || (cpu_y1->size[1] == 0)) {
            f_y = 0;
          } else {
            nx = cpu_y1->size[0];
            f_y = cpu_y1->size[1];
            if (nx >= f_y) {
              f_y = nx;
            }
          }
          i = expl_temp.x->size[0] * expl_temp.x->size[1];
          expl_temp.x->size[0] = 1;
          expl_temp.x->size[1] = f_y;
          emxEnsureCapacity_real_T(expl_temp.x, i, &m_emlrtRTEI);
          for (i = 0; i < f_y; i++) {
            expl_temp.x->data[i] = cpu_y1->data[i];
          }
        } else {
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              expl_temp.x, expl_temp.p, expl_temp.q,
              expl_temp.filterWithPadding, expl_temp.filterDelay);
        }
      } else {
        if (expl_temp.x->size[0] == 1) {
          boolean_T y1_needsGpuEnsureCapacity;
          dlast = std::ceil(expl_temp.p / expl_temp.q);
          xCh_outdatedOnCpu = false;
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = static_cast<int32_T>(dlast);
          cpu_y1->size[1] = expl_temp.x->size[1];
          emxEnsureCapacity_real_T(cpu_y1, i, &k_emlrtRTEI);
          y1_needsGpuEnsureCapacity = true;
          opts_dim = expl_temp.x->size[0] * expl_temp.x->size[1];
          if (opts_dim - 1 >= 0) {
            if ((static_cast<int32_T>(expl_temp.p) == 0) ||
                ((static_cast<int32_T>(expl_temp.p) > 0) &&
                 (static_cast<int32_T>(expl_temp.p) < 1))) {
              i3 = 1;
              i4 = 0;
            } else {
              i3 = static_cast<int32_T>(expl_temp.p);
              i4 = static_cast<int32_T>(expl_temp.p);
            }
            Ly = std::ceil(
                (0.0 * expl_temp.p +
                 static_cast<real_T>(expl_temp.filterWithPadding->size[0])) /
                expl_temp.q);
          }
          for (int32_T ii{0}; ii < opts_dim; ii++) {
            if (!(absx - 1.0 >= 0.0)) {
              b_y = nullptr;
              m = emlrtCreateCharArray(2, &iv1[0]);
              emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
              emlrtAssign(&b_y, m);
              c_y = nullptr;
              m1 = emlrtCreateDoubleScalar(absx - 1.0);
              emlrtAssign(&c_y, m1);
              emlrt_marshallIn(b_sprintf(b_y, c_y, &e_emlrtMCI),
                               "<output of sprintf>", unusedExpr);
            }
            i = yCol->size[0];
            yCol->size[0] = static_cast<int32_T>(absx);
            emxEnsureCapacity_real_T(yCol, i, &n_emlrtRTEI);
            for (i = 0; i < static_cast<int32_T>(absx); i++) {
              yCol->data[i] = 0.0;
            }
            f_y = div_s32(i4 - 1, i3) + 1;
            for (i = 0; i < f_y; i++) {
              yCol->data[i3 * i] = expl_temp.x->data[ii];
            }
            b_yCol = *yCol;
            b_expl_temp[0] = static_cast<int32_T>(absx);
            b_yCol.size = &b_expl_temp[0];
            b_yCol.numDimensions = 1;
            coder::b_conv2(&b_yCol, expl_temp.filterWithPadding, cpu_w,
                           &w_outdatedOnCpu, &gpu_w, &w_outdatedOnGpu);
            if (!(n - 1.0 >= 0.0)) {
              d_y = nullptr;
              m2 = emlrtCreateCharArray(2, &iv2[0]);
              emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
              emlrtAssign(&d_y, m2);
              e_y = nullptr;
              m3 = emlrtCreateDoubleScalar(n - 1.0);
              emlrtAssign(&e_y, m3);
              emlrt_marshallIn(b_sprintf(d_y, e_y, &e_emlrtMCI),
                               "<output of sprintf>", unusedExpr);
            }
            if (n == 0.0) {
              nx = 1;
              f_y = -1;
            } else {
              nx = static_cast<int32_T>(n);
              f_y = cpu_w->size[0] - 1;
            }
            i = b_cpu_y->size[0];
            b_cpu_y->size[0] = div_s32(f_y, nx) + 1;
            emxEnsureCapacity_real_T(b_cpu_y, i, &o_emlrtRTEI);
            i = f_y / nx;
            mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                    2147483647U);
            gpuEmxEnsureCapacity_real_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
            gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
            xCh_outdatedOnCpu = false;
            if (w_outdatedOnGpu) {
              gpuEmxMemcpyCpuToGpu_real_T(&gpu_w, cpu_w);
            }
            w_outdatedOnGpu = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              ec_cwt_kernel5<<<grid, block>>>(gpu_w, nx, i, b_gpu_y);
            }
            if (!(div_s32(f_y, nx) + 1 < Ly)) {
              i = b_cpu_y->size[0];
              if (Ly < 1.0) {
                b_cpu_y->size[0] = 0;
              } else {
                b_cpu_y->size[0] = static_cast<int32_T>(Ly);
              }
              emxEnsureCapacity_real_T(b_cpu_y, i, &p_emlrtRTEI);
              xCh_outdatedOnCpu = true;
            }
            if (std::isnan(dlast)) {
              i = c_cpu_y->size[0] * c_cpu_y->size[1];
              c_cpu_y->size[0] = 1;
              c_cpu_y->size[1] = 1;
              emxEnsureCapacity_real_T(c_cpu_y, i, &q_emlrtRTEI);
              gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
              y_needsGpuEnsureCapacity = false;
              ec_cwt_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(c_gpu_y);
            } else if (dlast < 1.0) {
              c_cpu_y->size[0] = 1;
              c_cpu_y->size[1] = 0;
            } else {
              i = c_cpu_y->size[0] * c_cpu_y->size[1];
              c_cpu_y->size[0] = 1;
              c_cpu_y->size[1] = static_cast<int32_T>(dlast - 1.0) + 1;
              emxEnsureCapacity_real_T(c_cpu_y, i, &q_emlrtRTEI);
              mwGetLaunchParameters1D(
                  computeNumIters(static_cast<int32_T>(dlast - 1.0)), &grid,
                  &block, 2147483647U);
              gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
              y_needsGpuEnsureCapacity = false;
              validLaunchParams = mwValidateLaunchParameters(grid, block);
              if (validLaunchParams) {
                ec_cwt_kernel8<<<grid, block>>>(
                    static_cast<int32_T>(dlast - 1.0), c_gpu_y);
              }
            }
            nx = cpu_y1->size[0];
            mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                    2147483647U);
            if (xCh_outdatedOnCpu) {
              gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
            }
            if (y_needsGpuEnsureCapacity) {
              gpuEmxEnsureCapacity_real_T(c_cpu_y, &c_gpu_y, true);
            }
            y_needsGpuEnsureCapacity = false;
            if (y1_needsGpuEnsureCapacity) {
              gpuEmxEnsureCapacity_real_T(cpu_y1, &gpu_y1, true);
            }
            y1_needsGpuEnsureCapacity = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              ec_cwt_kernel7<<<grid, block>>>(b_gpu_y, c_gpu_y,
                                              expl_temp.filterDelay, ii, nx - 1,
                                              gpu_y1, cpu_y1->size[0U]);
            }
            xCh_outdatedOnCpu = true;
          }
        } else {
          xCh_outdatedOnCpu = false;
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = expl_temp.x->size[0];
          cpu_y1->size[1] = expl_temp.x->size[1];
          emxEnsureCapacity_real_T(cpu_y1, i, &h_emlrtRTEI);
          for (i = 0; i < expl_temp.x->size[0] * expl_temp.x->size[1]; i++) {
            cpu_y1->data[i] = expl_temp.x->data[i];
          }
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_y1, expl_temp.p, expl_temp.q, expl_temp.filterWithPadding,
              expl_temp.filterDelay);
        }
        if ((expl_temp.x->size[0] == 1) || (expl_temp.x->size[1] == 1)) {
          nx = static_cast<int32_T>(expl_temp.inputSize[0]);
          f_y = cpu_y1->size[0];
          i = expl_temp.x->size[0] * expl_temp.x->size[1];
          expl_temp.x->size[0] = static_cast<int32_T>(expl_temp.inputSize[0]);
          expl_temp.x->size[1] = cpu_y1->size[0];
          emxEnsureCapacity_real_T(expl_temp.x, i, &l_emlrtRTEI);
          for (i = 0; i < nx * f_y; i++) {
            if (xCh_outdatedOnCpu) {
              gpuEmxMemcpyGpuToCpu_real_T(cpu_y1, &gpu_y1);
            }
            xCh_outdatedOnCpu = false;
            expl_temp.x->data[i] = cpu_y1->data[i];
          }
        } else {
          i = expl_temp.x->size[0] * expl_temp.x->size[1];
          expl_temp.x->size[0] = cpu_y1->size[1];
          expl_temp.x->size[1] = cpu_y1->size[0];
          emxEnsureCapacity_real_T(expl_temp.x, i, &l_emlrtRTEI);
          for (i = 0; i < cpu_y1->size[0]; i++) {
            for (nx = 0; nx < cpu_y1->size[1]; nx++) {
              if (xCh_outdatedOnCpu) {
                gpuEmxMemcpyGpuToCpu_real_T(cpu_y1, &gpu_y1);
              }
              xCh_outdatedOnCpu = false;
              expl_temp.x->data[nx + expl_temp.x->size[0] * i] =
                  cpu_y1->data[i + cpu_y1->size[0] * nx];
            }
          }
        }
      }
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = expl_temp.x->size[0];
      emxEnsureCapacity_real_T(y->data[ch].f1, i, &j_emlrtRTEI);
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = expl_temp.x->size[1];
      emxEnsureCapacity_real_T(y->data[ch].f1, i, &j_emlrtRTEI);
      for (i = 0; i < expl_temp.x->size[0] * expl_temp.x->size[1]; i++) {
        y->data[ch].f1->data[i] = expl_temp.x->data[i];
      }
      //  Downsample
    } else {
      // 'ec_cwt:42' else
      // 'ec_cwt:43' y{ch} = xCh;
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[0] = cpu_xCh->size[0];
      emxEnsureCapacity_real_T(y->data[ch].f1, i, &f_emlrtRTEI);
      i = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
      y->data[ch].f1->size[1] = cpu_xCh->size[1];
      emxEnsureCapacity_real_T(y->data[ch].f1, i, &f_emlrtRTEI);
      for (i = 0; i < cpu_xCh->size[0] * cpu_xCh->size[1]; i++) {
        if (xCh_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real_T(cpu_xCh, &gpu_xCh);
        }
        xCh_outdatedOnCpu = false;
        y->data[ch].f1->data[i] = cpu_xCh->data[i];
      }
    }
  }
  emxFree_real_T(&c_cpu_x);
  emxFreeStruct_ResampleParser(&expl_temp);
  emxFree_real_T(&c_cpu_y);
  emxFree_real_T(&b_cpu_y);
  emxFree_real_T(&cpu_w);
  emxFree_real_T(&yCol);
  emxFree_real_T(&cpu_y1);
  emxFree_real_T(&cpu_opts_x);
  emxFree_real_T(&cpu_xCh);
  emxFree_creal_T(&b_cpu_x);
  emxFree_real_T(&cpu_y);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_x);
  gpuEmxFree_real_T(&gpu_y);
  gpuEmxFree_creal_T(&c_gpu_x);
  gpuEmxFree_real_T(&gpu_xCh);
  gpuEmxFree_real_T(&gpu_opts_x);
  gpuEmxFree_real_T(&gpu_y1);
  gpuEmxFree_real_T(&gpu_w);
  gpuEmxFree_real_T(&b_gpu_y);
  gpuEmxFree_real_T(&c_gpu_y);
  gpuEmxFree_real_T(&b_gpu_x);
}

// End of code generation (ec_cwt.cu)

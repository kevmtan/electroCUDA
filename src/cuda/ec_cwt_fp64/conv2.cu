#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// conv2.cu
//
// Code generation for function 'conv2'
//

// Include files
#include "conv2.h"
#include "ec_cwt_data.h"
#include "ec_cwt_emxutil.h"
#include "ec_cwt_mexutil.h"
#include "ec_cwt_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo lf_emlrtRTEI{
    49,                                                               // lineNo
    31,                                                               // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo mf_emlrtRTEI{
    56,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo nf_emlrtRTEI{
    47,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo of_emlrtRTEI{
    75,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo pf_emlrtRTEI{
    49,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo qf_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "stencil_codegen", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "stencil_codegen.p" // pName
};

static emlrtRTEInfo rf_emlrtRTEI{
    172,                                                          // lineNo
    20,                                                           // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

static emlrtRTEInfo sf_emlrtRTEI{
    49,                                                               // lineNo
    13,                                                               // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo tf_emlrtRTEI{
    82,                                                           // lineNo
    9,                                                            // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

static emlrtRTEInfo
    uf_emlrtRTEI{
        158,      // lineNo
        24,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo vf_emlrtRTEI{
    1,                  // lineNo
    1,                  // colNo
    "gpu_conv2_kernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+images/gpu_conv2_kernel.p" // pName
};

// Function Declarations
namespace coder {
static void
b_conv2NonSeparable(emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
                    emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
                    emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
                    emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
                    emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu);

static void
conv2NonSeparable(emxArray_real_T *cpu_a, emxArray_real_T *gpu_a,
                  boolean_T *a_outdatedOnGpu, emxArray_real_T *cpu_b,
                  emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
                  emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
                  emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu);

} // namespace coder
static __global__ void conv2NonSeparable_kernel43(const int32_T b,
                                                  emxArray_real_T c);

static __global__ void conv2NonSeparable_kernel44(const int32_T b,
                                                  emxArray_real_T c);

static __global__ void conv2NonSeparable_kernel45(const emxArray_real_T c,
                                                  const int32_T b,
                                                  emxArray_real_T b_c);

static __global__ void conv2NonSeparable_kernel46(const int32_T OH,
                                                  emxArray_real_T expanded);

static __global__ void conv2NonSeparable_kernel47(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel48(const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv1);

static __global__ void conv2NonSeparable_kernel49(
    const emxArray_real_T a, const int32_T iv_dim0, const emxArray_int32_T iv1,
    const emxArray_int32_T iv, const int32_T b_iv, const int32_T c_iv,
    emxArray_real_T expanded, int32_T expanded_dim0);

static __global__ void conv2NonSeparable_kernel50(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_real_T b, const int32_T c, const int32_T d,
    emxArray_real_T b_c, int32_T b_dim0, int32_T expanded_dim0, int32_T c_dim0);

static __global__ void conv2NonSeparable_kernel51(const int32_T b,
                                                  emxArray_real_T c);

static __global__ void conv2NonSeparable_kernel52(const int32_T OH,
                                                  emxArray_real_T expanded);

static __global__ void conv2NonSeparable_kernel53(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel54(const emxArray_real_T a,
                                                  const emxArray_int32_T iv,
                                                  const int32_T b,
                                                  emxArray_real_T expanded);

static __global__ void
conv2NonSeparable_kernel55(const emxArray_real_T expanded,
                           const emxArray_int32_T rows, const emxArray_real_T b,
                           const int32_T c, emxArray_real_T b_c,
                           int32_T b_dim0);

static __global__ void conv2_kernel28(const emxArray_real_T a,
                                      const int32_T b_a, emxArray_real_T c_a,
                                      int32_T a_dim0);

static __global__ void conv2_kernel29(const int32_T a, emxArray_real_T c);

static __global__ void conv2_kernel30(const real_T b, const emxArray_real_T a,
                                      emxArray_real_T c);

static __global__ void conv2_kernel31(const emxArray_real_T c,
                                      const int32_T b_c, emxArray_real_T c_c,
                                      int32_T c_dim0);

static __global__ void conv2_kernel32(const int32_T a,
                                      emxArray_real_T expanded);

static __global__ void conv2_kernel33(const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel34(const emxArray_real_T a,
                                      const emxArray_int32_T iv,
                                      const int32_T b,
                                      emxArray_real_T expanded);

static __global__ void conv2_kernel35(const real_T b,
                                      const emxArray_real_T expanded,
                                      const int32_T c, emxArray_real_T b_c);

static __global__ void conv2_kernel36(const int32_T b, emxArray_real_T c);

static __global__ void conv2_kernel37(const int32_T b, emxArray_real_T c);

static __global__ void conv2_kernel38(const emxArray_real_T a,
                                      const emxArray_real_T b,
                                      emxArray_real_T c);

static __global__ void conv2_kernel39(const int32_T OH,
                                      emxArray_real_T expanded);

static __global__ void conv2_kernel40(const int32_T offsetH,
                                      const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv1);

static __global__ void conv2_kernel41(const emxArray_real_T b, const int32_T mc,
                                      const emxArray_int32_T iv1,
                                      const int32_T c, emxArray_real_T expanded,
                                      int32_T expanded_dim0);

static __global__ void conv2_kernel42(const emxArray_real_T expanded,
                                      const emxArray_int32_T cols,
                                      const emxArray_int32_T rows,
                                      const emxArray_real_T a, const int32_T b,
                                      const int32_T c, emxArray_real_T b_c,
                                      int32_T a_dim0, int32_T a_dim1,
                                      int32_T expanded_dim0, int32_T c_dim0);

static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line);

// Function Definitions
//
//
namespace coder {
static void
b_conv2NonSeparable(emxArray_real_T *cpu_a, boolean_T *a_outdatedOnCpu,
                    emxArray_real_T *gpu_a, boolean_T *a_outdatedOnGpu,
                    emxArray_real_T *cpu_b, boolean_T *b_outdatedOnCpu,
                    emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
                    emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *cpu_expanded;
  int32_T k;
  int32_T mc;
  boolean_T validLaunchParams;
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_expanded);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  k = cpu_c->size[0];
  cpu_c->size[0] = mc;
  emxEnsureCapacity_real_T(cpu_c, k, &of_emlrtRTEI);
  mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    conv2NonSeparable_kernel51<<<grid, block>>>(mc - 1, *gpu_c);
  }
  *c_outdatedOnGpu = false;
  *c_outdatedOnCpu = true;
  emxInit_real_T(&cpu_expanded, 1, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &tf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &qf_emlrtRTEI, true);
  if ((cpu_a->size[0] != 0) && (cpu_b->size[0] != 0)) {
    if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
      real_T blockDims_idx_0;
      if (*b_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      blockDims_idx_0 = cpu_b->data[0];
      k = cpu_c->size[0];
      cpu_c->size[0] = 1;
      emxEnsureCapacity_real_T(cpu_c, k, &of_emlrtRTEI);
      if (*a_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_a, gpu_a);
      }
      *a_outdatedOnCpu = false;
      cpu_c->data[0] = cpu_a->data[0] * blockDims_idx_0;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      real_T blockDims_idx_0;
      int32_T b_OH;
      int32_T n;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims_idx_0;
      OH = (static_cast<uint32_T>(cpu_a->size[0]) +
            static_cast<uint32_T>(cpu_b->size[0])) -
           1U;
      mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      blockDims_idx_0 = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
          static_cast<real_T>(mc));
      threadDims_idx_0 = static_cast<int8_T>(mc);
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
      if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_b->size[0];
      }
      k = cpu_expanded->size[0];
      cpu_expanded->size[0] = mc - 1;
      emxEnsureCapacity_real_T(cpu_expanded, k, &qf_emlrtRTEI);
      mc = (static_cast<int32_T>(OH) + cpu_b->size[0]) - 2;
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_expanded, &gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel52<<<grid, block>>>(mc, gpu_expanded);
      }
      n = cpu_a->size[0];
      k = cpu_y->size[0] * cpu_y->size[1];
      cpu_y->size[0] = 1;
      cpu_y->size[1] = cpu_a->size[0];
      emxEnsureCapacity_int32_T(cpu_y, k, &rf_emlrtRTEI);
      cpu_y->data[0] = 1;
      mc = 1;
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_y->data[k + 1] = mc;
      }
      k = cpu_iv->size[0];
      cpu_iv->size[0] = cpu_y->size[1];
      emxEnsureCapacity_int32_T(cpu_iv, k, &qf_emlrtRTEI);
      mc = cpu_y->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
      gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel53<<<grid, block>>>(offsetH, gpu_y, mc, gpu_iv);
      }
      mc = cpu_iv->size[0];
      mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
      if (*a_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
      }
      *a_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel54<<<grid, block>>>(*gpu_a, gpu_iv, mc - 1,
                                                    gpu_expanded);
      }
      n = cpu_b->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_b->size[0];
      emxEnsureCapacity_int32_T(cpu_rows, k, &rf_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      k = cpu_c->size[0];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      emxEnsureCapacity_real_T(cpu_c, k, &qf_emlrtRTEI);
      if (blockDims_idx_0 < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims_idx_0);
      } else {
        OH = MAX_uint32_T;
      }
      mwApplyLaunchParameters(
          computeNumIters(b_OH - 1), dim3(OH, 1U, 1U),
          dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid, &block);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel55<<<grid, block>>>(
            gpu_expanded, gpu_rows, *gpu_b, b_OH - 1, *gpu_c, cpu_b->size[0U]);
      }
    }
  }
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&cpu_y);
  emxFree_real_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_rows);
}

//
//
static void
conv2NonSeparable(emxArray_real_T *cpu_a, emxArray_real_T *gpu_a,
                  boolean_T *a_outdatedOnGpu, emxArray_real_T *cpu_b,
                  emxArray_real_T *gpu_b, boolean_T *b_outdatedOnGpu,
                  emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
                  emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T b_gpu_c;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *cpu_expanded;
  int32_T k;
  int32_T mc;
  int32_T nc;
  boolean_T validLaunchParams;
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_expanded);
  gpuEmxReset_real_T(&b_gpu_c);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  if (cpu_a->size[1] == 0) {
    nc = 1;
  } else {
    nc = cpu_a->size[1];
  }
  k = cpu_c->size[0] * cpu_c->size[1];
  cpu_c->size[0] = mc;
  cpu_c->size[1] = nc;
  emxEnsureCapacity_real_T(cpu_c, k, &of_emlrtRTEI);
  k = mc * nc - 1;
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    conv2NonSeparable_kernel43<<<grid, block>>>(k, *gpu_c);
  }
  emxInit_real_T(&b_cpu_c, 1, &vf_emlrtRTEI, true);
  emxInit_real_T(&cpu_expanded, 2, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &tf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &qf_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &tf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 1, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &qf_emlrtRTEI, true);
  if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0) || (cpu_b->size[0] == 0)) {
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_real_T(cpu_c, k, &of_emlrtRTEI);
    k = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel44<<<grid, block>>>(k, *gpu_c);
    }
  } else if ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1) &&
             (cpu_b->size[0] == 1)) {
    real_T alpha1;
    real_T beta1;
    k = b_cpu_c->size[0];
    b_cpu_c->size[0] = 1;
    emxEnsureCapacity_real_T(b_cpu_c, k, &uf_emlrtRTEI);
    alpha1 = 1.0;
    beta1 = 0.0;
    gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
    if (*a_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    cublasCheck(hipblasDgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            1, 1, 1, (double *)&alpha1,
                            (double *)&gpu_a->data[0], 1,
                            (double *)&gpu_b->data[0], 1, (double *)&beta1,
                            (double *)&b_gpu_c.data[0], 1),
                __FILE__, __LINE__);
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_real_T(cpu_c, k, &of_emlrtRTEI);
    k = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel45<<<grid, block>>>(b_gpu_c, k, *gpu_c);
    }
  } else {
    real_T blockDims[3];
    int32_T iv[2];
    int32_T OW;
    int32_T b_OH;
    int32_T offsetH;
    uint32_T OH;
    int8_T threadDims[3];
    OH = (static_cast<uint32_T>(cpu_a->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    nc = static_cast<int32_T>(
        std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
    blockDims[0] =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
                   static_cast<real_T>(mc));
    blockDims[1] =
        std::floor((static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                                        static_cast<uint32_T>(nc)) -
                    1.0) /
                   static_cast<real_T>(nc));
    threadDims[0] = static_cast<int8_T>(mc);
    threadDims[1] = static_cast<int8_T>(nc);
    OW = cpu_a->size[1];
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    b_OH = static_cast<int32_T>(OH);
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
      mc = MAX_int32_T;
    } else {
      mc = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[0] = mc - 1;
    emxEnsureCapacity_real_T(cpu_expanded, k, &qf_emlrtRTEI);
    if (cpu_a->size[1] > 2147483646) {
      mc = MAX_int32_T;
    } else {
      mc = cpu_a->size[1] + 1;
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[1] = mc - 1;
    emxEnsureCapacity_real_T(cpu_expanded, k, &qf_emlrtRTEI);
    k = ((static_cast<int32_T>(OH) + cpu_b->size[0]) - 1) * cpu_a->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel46<<<grid, block>>>(k, gpu_expanded);
    }
    nc = cpu_a->size[0];
    k = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_a->size[0];
    emxEnsureCapacity_int32_T(cpu_y, k, &rf_emlrtRTEI);
    cpu_y->data[0] = 1;
    mc = 1;
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_y->data[k + 1] = mc;
    }
    k = cpu_iv->size[0];
    cpu_iv->size[0] = cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv, k, &qf_emlrtRTEI);
    k = cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel47<<<grid, block>>>(offsetH, gpu_y, k, gpu_iv);
    }
    nc = cpu_a->size[1];
    k = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_a->size[1];
    emxEnsureCapacity_int32_T(b_cpu_y, k, &rf_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    mc = 1;
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      b_cpu_y->data[k + 1] = mc;
    }
    k = cpu_iv1->size[0];
    cpu_iv1->size[0] = b_cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv1, k, &qf_emlrtRTEI);
    k = b_cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel48<<<grid, block>>>(b_gpu_y, k, gpu_iv1);
    }
    iv[0] = cpu_iv->size[0];
    iv[1] = cpu_iv1->size[0];
    mwGetLaunchParameters1D(computeNumIters(iv[1] - 1, iv[0] - 1), &grid,
                            &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    if (*a_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel49<<<grid, block>>>(
          *gpu_a, iv[0], gpu_iv1, gpu_iv, iv[0] - 1, iv[1] - 1, gpu_expanded,
          cpu_expanded->size[0U]);
    }
    nc = cpu_b->size[0];
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = cpu_b->size[0];
    emxEnsureCapacity_int32_T(cpu_rows, k, &rf_emlrtRTEI);
    cpu_rows->data[0] = 0;
    mc = 0;
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_rows->data[k + 1] = mc;
    }
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = static_cast<int32_T>(OH);
    cpu_c->size[1] = cpu_a->size[1];
    emxEnsureCapacity_real_T(cpu_c, k, &qf_emlrtRTEI);
    if (blockDims[0] < 4.294967296E+9) {
      OH = static_cast<uint32_T>(blockDims[0]);
    } else {
      OH = MAX_uint32_T;
    }
    mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                            dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                            dim3(static_cast<uint32_T>(threadDims[0]),
                                 static_cast<uint32_T>(threadDims[1]), 1U),
                            &grid, &block);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel50<<<grid, block>>>(
          gpu_expanded, gpu_rows, *gpu_b, b_OH - 1, OW - 1, *gpu_c,
          cpu_b->size[0U], cpu_expanded->size[0U], cpu_c->size[0U]);
    }
  }
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv1);
  emxFree_int32_T(&b_cpu_y);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&cpu_y);
  emxFree_real_T(&cpu_expanded);
  emxFree_real_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&b_gpu_c);
  gpuEmxFree_real_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_int32_T(&gpu_rows);
  *c_outdatedOnCpu = true;
  *c_outdatedOnGpu = false;
}

} // namespace coder
static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel43(
    const int32_T b, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    c.data[i2] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel44(
    const int32_T b, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    c.data[i2] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel45(
    const emxArray_real_T c, const int32_T b, emxArray_real_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    b_c.data[i2] = c.data[i2];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel46(
    const int32_T OH, emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    expanded.data[i2] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel47(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    int32_T q1;
    i2 = static_cast<int32_T>(idx);
    q1 = y.data[i2];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[i2] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel48(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    iv1.data[i2] = y.data[i2] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel49(
    const emxArray_real_T a, const int32_T iv_dim0, const emxArray_int32_T iv1,
    const emxArray_int32_T iv, const int32_T b_iv, const int32_T c_iv,
    emxArray_real_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_iv) + 1UL) *
                (static_cast<uint64_T>(b_iv) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    int32_T ocol;
    ocol = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_iv) + 1UL));
    i2 = static_cast<int32_T>((idx - static_cast<uint64_T>(ocol)) /
                              (static_cast<uint64_T>(b_iv) + 1UL));
    expanded.data[iv.data[ocol] + expanded_dim0 * iv1.data[i2]] =
        a.data[ocol + iv_dim0 * i2];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel50(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_real_T b, const int32_T c, const int32_T d,
    emxArray_real_T b_c, int32_T b_dim0, int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(d) + 1UL) * (static_cast<uint64_T>(c) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T ocol;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(c) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(c) + 1UL));
    b_cv = 0.0;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[(q1 + expanded_dim0 * ocol) - 1] *
              b.data[(b_dim0 - m) - 1];
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel51(
    const int32_T b, emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel52(
    const int32_T OH, emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel53(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel54(
    const emxArray_real_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv.data[orow]] = a.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel55(
    const emxArray_real_T expanded, const emxArray_int32_T rows,
    const emxArray_real_T b, const int32_T c, emxArray_real_T b_c,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_cv = 0.0;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[q1 - 1] * b.data[(b_dim0 - m) - 1];
    }
    b_c.data[orow] = b_cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel28(
    const emxArray_real_T a, const int32_T b_a, emxArray_real_T c_a,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c_a.data[ocol] = a.data[a_dim0 * ocol];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel29(const int32_T a,
                                                           emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel30(
    const real_T b, const emxArray_real_T a, emxArray_real_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = a.data[0] * b;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel31(
    const emxArray_real_T c, const int32_T b_c, emxArray_real_T c_c,
    int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c_c.data[c_dim0 * ocol] = c.data[ocol];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel32(const int32_T a,
                                                   emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel33(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    iv.data[ocol] = y.data[ocol] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel34(
    const emxArray_real_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[iv.data[ocol]] = a.data[ocol];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel35(
    const real_T b, const emxArray_real_T expanded, const int32_T c,
    emxArray_real_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_c.data[orow] = expanded.data[orow] * b;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel36(const int32_T b,
                                                           emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel37(const int32_T b,
                                                           emxArray_real_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel38(
    const emxArray_real_T a, const emxArray_real_T b, emxArray_real_T c)
{
  int32_T k;
  k = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (k < 1) {
    real_T b_cv;
    b_cv = b.data[0] * a.data[0];
    c.data[0] = b_cv;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel39(const int32_T OH,
                                                   emxArray_real_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[ocol] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel40(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    int32_T q1;
    ocol = static_cast<int32_T>(idx);
    q1 = y.data[ocol];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[ocol] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel41(
    const emxArray_real_T b, const int32_T mc, const emxArray_int32_T iv1,
    const int32_T c, emxArray_real_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[iv1.data[ocol] + expanded_dim0 * mc] = b.data[ocol];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel42(
    const emxArray_real_T expanded, const emxArray_int32_T cols,
    const emxArray_int32_T rows, const emxArray_real_T a, const int32_T b,
    const int32_T c, emxArray_real_T b_c, int32_T a_dim0, int32_T a_dim1,
    int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T b_cv;
    int32_T ocol;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_cv = 0.0;
    for (int32_T n{0}; n < a_dim1; n++) {
      for (int32_T m{0}; m < a_dim0; m++) {
        int32_T q1;
        int32_T qY;
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] *
                a.data[((a_dim0 - m) + a_dim0 * ((a_dim1 - n) - 1)) - 1];
      }
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

//
//
namespace coder {
void b_conv2(const emxArray_real_T *cpu_a, const emxArray_real_T *cpu_b,
             emxArray_real_T *cpu_c, boolean_T *c_outdatedOnCpu,
             emxArray_real_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_b;
  boolean_T a_outdatedOnCpu;
  boolean_T a_outdatedOnGpu;
  boolean_T b_outdatedOnCpu;
  boolean_T b_outdatedOnGpu;
  gpuEmxReset_real_T(&gpu_b);
  gpuEmxReset_real_T(&gpu_a);
  b_outdatedOnCpu = false;
  b_outdatedOnGpu = true;
  a_outdatedOnCpu = false;
  a_outdatedOnGpu = true;
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    int32_T i;
    i = cpu_c->size[0];
    cpu_c->size[0] = 1;
    emxEnsureCapacity_real_T(cpu_c, i, &pf_emlrtRTEI);
    cpu_c->data[0] = cpu_a->data[0] * cpu_b->data[0];
    *c_outdatedOnCpu = false;
    *c_outdatedOnGpu = true;
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    b_conv2NonSeparable((emxArray_real_T *)cpu_b, &b_outdatedOnCpu, &gpu_b,
                        &b_outdatedOnGpu, (emxArray_real_T *)cpu_a,
                        &a_outdatedOnCpu, &gpu_a, &a_outdatedOnGpu, cpu_c,
                        c_outdatedOnCpu, gpu_c, c_outdatedOnGpu);
  } else {
    b_conv2NonSeparable((emxArray_real_T *)cpu_a, &a_outdatedOnCpu, &gpu_a,
                        &a_outdatedOnGpu, (emxArray_real_T *)cpu_b,
                        &b_outdatedOnCpu, &gpu_b, &b_outdatedOnGpu, cpu_c,
                        c_outdatedOnCpu, gpu_c, c_outdatedOnGpu);
  }
  gpuEmxFree_real_T(&gpu_a);
  gpuEmxFree_real_T(&gpu_b);
}

//
//
void conv2(const emxArray_real_T *cpu_a, emxArray_real_T *cpu_b,
           boolean_T *b_outdatedOnCpu, emxArray_real_T *gpu_b,
           boolean_T *b_outdatedOnGpu, emxArray_real_T *cpu_c,
           boolean_T *c_outdatedOnCpu, emxArray_real_T *gpu_c,
           boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_cols;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_cols;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real_T b_gpu_a;
  emxArray_real_T b_gpu_c;
  emxArray_real_T b_gpu_expanded;
  emxArray_real_T gpu_a;
  emxArray_real_T gpu_expanded;
  emxArray_real_T *b_cpu_a;
  emxArray_real_T *b_cpu_c;
  emxArray_real_T *b_cpu_expanded;
  emxArray_real_T *cpu_expanded;
  boolean_T a_outdatedOnGpu;
  gpuEmxReset_int32_T(&gpu_cols);
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_real_T(&b_gpu_expanded);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_real_T(&gpu_expanded);
  gpuEmxReset_real_T(&b_gpu_a);
  gpuEmxReset_real_T(&b_gpu_c);
  gpuEmxReset_real_T(&gpu_a);
  a_outdatedOnGpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real_T(&b_cpu_c, 1, &sf_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_a, 1, &lf_emlrtRTEI, true);
  emxInit_real_T(&cpu_expanded, 1, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &tf_emlrtRTEI, true);
  emxInit_real_T(&b_cpu_expanded, 2, &qf_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &tf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 1, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &qf_emlrtRTEI, true);
  emxInit_int32_T(&cpu_cols, 2, &qf_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    if (cpu_a->size[1] < 1) {
      int32_T k;
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      emxEnsureCapacity_real_T(cpu_c, k, &nf_emlrtRTEI);
      cpu_c->data[0] = 0.0;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      real_T b;
      int32_T k;
      boolean_T validLaunchParams;
      k = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[1];
      emxEnsureCapacity_real_T(b_cpu_a, k, &lf_emlrtRTEI);
      k = cpu_a->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_real_T(b_cpu_a, &b_gpu_a, true);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel28<<<grid, block>>>(gpu_a, k, b_gpu_a, cpu_a->size[0U]);
      }
      if (*b_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      b = cpu_b->data[0];
      k = b_cpu_c->size[0];
      b_cpu_c->size[0] = b_cpu_a->size[0];
      emxEnsureCapacity_real_T(b_cpu_c, k, &of_emlrtRTEI);
      k = b_cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel29<<<grid, block>>>(k, b_gpu_c);
      }
      if (b_cpu_a->size[0] == 1) {
        b = cpu_b->data[0];
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = 1;
        emxEnsureCapacity_real_T(b_cpu_c, k, &of_emlrtRTEI);
        gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
        conv2_kernel30<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b, b_gpu_a,
                                                                b_gpu_c);
      } else {
        int32_T b_OH;
        int32_T mc;
        int32_T nc;
        int32_T offsetH;
        int8_T threadDims[3];
        nc = static_cast<int32_T>(std::fmin(
            32.0, (static_cast<real_T>(b_cpu_a->size[0]) + 1.0) - 1.0));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(b_cpu_a->size[0]) +
                                 static_cast<uint32_T>(nc)) -
             1.0) /
            static_cast<real_T>(nc)));
        threadDims[0] = static_cast<int8_T>(nc);
        b_OH = b_cpu_a->size[0];
        if (b_cpu_a->size[0] > 2147483646) {
          mc = MAX_int32_T;
        } else {
          mc = b_cpu_a->size[0] + 1;
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        emxEnsureCapacity_real_T(cpu_expanded, k, &qf_emlrtRTEI);
        k = b_cpu_a->size[0] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_real_T(cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel32<<<grid, block>>>(k, gpu_expanded);
        }
        nc = b_cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = b_cpu_a->size[0];
        emxEnsureCapacity_int32_T(cpu_y, k, &rf_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        for (k = 0; k <= nc - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        emxEnsureCapacity_int32_T(cpu_iv, k, &qf_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel33<<<grid, block>>>(gpu_y, k, gpu_iv);
        }
        nc = cpu_iv->size[0];
        mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel34<<<grid, block>>>(b_gpu_a, gpu_iv, nc - 1,
                                          gpu_expanded);
        }
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = b_cpu_a->size[0];
        emxEnsureCapacity_real_T(b_cpu_c, k, &qf_emlrtRTEI);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims[0]), 1U, 1U), &grid, &block);
        gpuEmxEnsureCapacity_real_T(b_cpu_c, &b_gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel35<<<grid, block>>>(b, gpu_expanded, b_OH - 1, b_gpu_c);
        }
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = b_cpu_c->size[0];
      emxEnsureCapacity_real_T(cpu_c, k, &pf_emlrtRTEI);
      k = b_cpu_c->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel31<<<grid, block>>>(b_gpu_c, k, *gpu_c, cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    int32_T nc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    if (cpu_a->size[1] == 0) {
      nc = 1;
    } else {
      nc = cpu_a->size[1];
    }
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_real_T(cpu_c, k, &mf_emlrtRTEI);
    k = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2_kernel36<<<grid, block>>>(k, *gpu_c);
    }
    if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0)) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = nc;
      emxEnsureCapacity_real_T(cpu_c, k, &mf_emlrtRTEI);
      k = mc * nc - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel37<<<grid, block>>>(k, *gpu_c);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else if ((cpu_b->size[0] == 1) &&
               ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1))) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      emxEnsureCapacity_real_T(cpu_c, k, &mf_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      conv2_kernel38<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_a, *gpu_b,
                                                              *gpu_c);
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else {
      real_T blockDims[3];
      int32_T OW;
      int32_T b_OH;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims[3];
      OH = (static_cast<uint32_T>(cpu_b->size[0]) +
            static_cast<uint32_T>(cpu_a->size[0])) -
           1U;
      nc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      mc = static_cast<int32_T>(
          std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
      blockDims[0] = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(nc) - 1.0)) /
          static_cast<real_T>(nc));
      blockDims[1] = std::floor(
          (static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                               static_cast<uint32_T>(mc)) -
           1.0) /
          static_cast<real_T>(mc));
      threadDims[0] = static_cast<int8_T>(nc);
      threadDims[1] = static_cast<int8_T>(mc);
      OW = cpu_a->size[1];
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
      if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_a->size[0];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[0] = mc - 1;
      emxEnsureCapacity_real_T(b_cpu_expanded, k, &qf_emlrtRTEI);
      if (cpu_a->size[1] > MAX_int32_T - cpu_a->size[1]) {
        mc = MAX_int32_T;
      } else {
        mc = cpu_a->size[1] + cpu_a->size[1];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[1] = mc - 1;
      emxEnsureCapacity_real_T(b_cpu_expanded, k, &qf_emlrtRTEI);
      k = ((static_cast<int32_T>(OH) + cpu_a->size[0]) - 1) *
              ((cpu_a->size[1] + cpu_a->size[1]) - 1) -
          1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(b_cpu_expanded, &b_gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel39<<<grid, block>>>(k, b_gpu_expanded);
      }
      nc = cpu_b->size[0];
      k = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = 1;
      b_cpu_y->size[1] = cpu_b->size[0];
      emxEnsureCapacity_int32_T(b_cpu_y, k, &rf_emlrtRTEI);
      b_cpu_y->data[0] = 1;
      mc = 1;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        b_cpu_y->data[k + 1] = mc;
      }
      k = cpu_iv1->size[0];
      cpu_iv1->size[0] = b_cpu_y->size[1];
      emxEnsureCapacity_int32_T(cpu_iv1, k, &qf_emlrtRTEI);
      k = b_cpu_y->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel40<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv1);
      }
      mc = static_cast<int32_T>(
               std::floor(static_cast<real_T>(cpu_a->size[1]) / 2.0)) +
           static_cast<int32_T>(
               std::floor((static_cast<real_T>(cpu_a->size[1]) - 1.0) / 2.0));
      nc = cpu_iv1->size[0];
      mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel41<<<grid, block>>>(*gpu_b, mc, gpu_iv1, nc - 1,
                                        b_gpu_expanded,
                                        b_cpu_expanded->size[0U]);
      }
      nc = cpu_a->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_a->size[0];
      emxEnsureCapacity_int32_T(cpu_rows, k, &rf_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      nc = cpu_a->size[1];
      k = cpu_cols->size[0] * cpu_cols->size[1];
      cpu_cols->size[0] = 1;
      cpu_cols->size[1] = cpu_a->size[1];
      emxEnsureCapacity_int32_T(cpu_cols, k, &rf_emlrtRTEI);
      cpu_cols->data[0] = 0;
      mc = 0;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_cols->data[k + 1] = mc;
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      cpu_c->size[1] = cpu_a->size[1];
      emxEnsureCapacity_real_T(cpu_c, k, &qf_emlrtRTEI);
      if (blockDims[0] < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims[0]);
      } else {
        OH = MAX_uint32_T;
      }
      mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                              dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                              dim3(static_cast<uint32_T>(threadDims[0]),
                                   static_cast<uint32_T>(threadDims[1]), 1U),
                              &grid, &block);
      gpuEmxEnsureCapacity_int32_T(cpu_cols, &gpu_cols, false);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      gpuEmxEnsureCapacity_real_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_real_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_cols, cpu_cols);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel42<<<grid, block>>>(
            b_gpu_expanded, gpu_cols, gpu_rows, gpu_a, b_OH - 1, OW - 1, *gpu_c,
            cpu_a->size[0U], cpu_a->size[1U], b_cpu_expanded->size[0U],
            cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else {
    conv2NonSeparable((emxArray_real_T *)cpu_a, &gpu_a, &a_outdatedOnGpu, cpu_b,
                      gpu_b, b_outdatedOnGpu, cpu_c, c_outdatedOnCpu, gpu_c,
                      c_outdatedOnGpu);
  }
  emxFree_int32_T(&cpu_cols);
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv1);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&b_cpu_y);
  emxFree_real_T(&b_cpu_expanded);
  emxFree_int32_T(&cpu_y);
  emxFree_real_T(&cpu_expanded);
  emxFree_real_T(&b_cpu_a);
  emxFree_real_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_a);
  gpuEmxFree_real_T(&b_gpu_c);
  gpuEmxFree_real_T(&b_gpu_a);
  gpuEmxFree_real_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_real_T(&b_gpu_expanded);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_int32_T(&gpu_rows);
  gpuEmxFree_int32_T(&gpu_cols);
}

} // namespace coder

// End of code generation (conv2.cu)

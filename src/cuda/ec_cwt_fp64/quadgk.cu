#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// quadgk.cu
//
// Code generation for function 'quadgk'
//

// Include files
#include "quadgk.h"
#include "ec_cwt_data.h"
#include "ec_cwt_emxutil.h"
#include "ec_cwt_mexutil.h"
#include "ec_cwt_types.h"
#include "morseproperties.h"
#include "rt_nonfinite.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtRTEInfo qc_emlrtRTEI{
    63,                // lineNo
    57,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    63,                // lineNo
    20,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo sc_emlrtRTEI{
    1,                                                                // lineNo
    23,                                                               // colNo
    "quadgk",                                                         // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/funfun/quadgk.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    63,                // lineNo
    34,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    63,                // lineNo
    16,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    63,                // lineNo
    60,                // colNo
    "morseproperties", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "morseproperties.m" // pName
};

// Function Definitions
void binary_expand_op_3(emxArray_real_T *in1, const emxArray_real_T *in2,
                        const emxArray_real_T *in3, const emxArray_real_T *in4)
{
  int32_T i;
  int32_T stride_0_1;
  int32_T stride_1_1;
  i = in1->size[0] * in1->size[1];
  in1->size[0] = 1;
  in1->size[1] = in4->size[1];
  emxEnsureCapacity_real_T(in1, i, &jc_emlrtRTEI);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_1 = (in3->size[1] != 1);
  for (i = 0; i < in4->size[1]; i++) {
    in1->data[i] =
        in2->data[i * stride_0_1] * in3->data[i * stride_1_1] * in4->data[i];
  }
}

//
//
namespace coder {
real_T quadgk(real_T fun_workspace_be, real_T fun_workspace_ga)
{
  emxArray_real_T *a;
  emxArray_real_T *b_dv1;
  emxArray_real_T *b_x;
  emxArray_real_T *b_y;
  emxArray_real_T *c_y;
  emxArray_real_T *dv4;
  emxArray_real_T *fx;
  emxArray_real_T *x;
  emxArray_real_T *xt;
  emxArray_real_T *y;
  real_T subs[1298];
  real_T interval[650];
  real_T errsub[649];
  real_T qsub[649];
  real_T pathlen;
  real_T q;
  int32_T ix;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  interval[0] = 0.0;
  interval[1] = 1.0;
  std::memset(&interval[2], 0, 648U * sizeof(real_T));
  q = 0.0;
  ix = split(interval, 2, &pathlen);
  if (!(pathlen > 0.0)) {
    real_T midpt;
    midpt = fun_workspace_be * rt_powd_snf(rtInf, fun_workspace_be - 1.0) -
            fun_workspace_ga *
                rt_powd_snf(rtInf, (fun_workspace_be + fun_workspace_ga) - 1.0);
    q = rtInf *
        (midpt * midpt * std::exp(-2.0 * rt_powd_snf(rtInf, fun_workspace_ga)));
  } else {
    real_T err_ok;
    real_T q_ok;
    int32_T k;
    int32_T nsubs;
    boolean_T first_iteration;
    nsubs = ix - 2;
    for (k = 0; k <= nsubs; k++) {
      subs[k << 1] = interval[k];
      subs[(k << 1) + 1] = interval[k + 1];
    }
    q_ok = 0.0;
    err_ok = 0.0;
    first_iteration = true;
    emxInit_real_T(&x, 2, &ec_emlrtRTEI, true);
    emxInit_real_T(&b_x, 2, &kc_emlrtRTEI, true);
    emxInit_real_T(&xt, 2, &lc_emlrtRTEI, true);
    emxInit_real_T(&y, 2, &rc_emlrtRTEI, true);
    emxInit_real_T(&fx, 2, &sc_emlrtRTEI, true);
    emxInit_real_T(&b_y, 2, &tc_emlrtRTEI, true);
    emxInit_real_T(&a, 2, &pc_emlrtRTEI, true);
    emxInit_real_T(&b_dv1, 2, &uc_emlrtRTEI, true);
    emxInit_real_T(&c_y, 2, &vc_emlrtRTEI, true);
    emxInit_real_T(&dv4, 2, &uc_emlrtRTEI, true);
    int32_T exitg1;
    do {
      real_T d;
      real_T halfh;
      real_T midpt;
      real_T tau;
      boolean_T guard1;
      exitg1 = 0;
      ix = x->size[0] * x->size[1];
      x->size[0] = 1;
      x->size[1] = 15 * (nsubs + 1);
      emxEnsureCapacity_real_T(x, ix, &ec_emlrtRTEI);
      ix = -1;
      for (k = 0; k <= nsubs; k++) {
        d = subs[k << 1];
        tau = subs[(k << 1) + 1];
        midpt = (d + tau) / 2.0;
        halfh = (tau - d) / 2.0;
        for (int32_T j{0}; j < 15; j++) {
          x->data[(ix + j) + 1] = dv[j] * halfh + midpt;
        }
        ix += 15;
      }
      ix = b_x->size[0] * b_x->size[1];
      b_x->size[0] = 1;
      b_x->size[1] = x->size[1];
      emxEnsureCapacity_real_T(b_x, ix, &fc_emlrtRTEI);
      ix = xt->size[0] * xt->size[1];
      xt->size[0] = 1;
      xt->size[1] = x->size[1];
      emxEnsureCapacity_real_T(xt, ix, &gc_emlrtRTEI);
      ix = x->size[1];
      for (k = 0; k < ix; k++) {
        d = x->data[k];
        midpt = d / (1.0 - d);
        b_x->data[k] = midpt * midpt;
        xt->data[k] = 2.0 * midpt / ((1.0 - d) * (1.0 - d));
      }
      guard1 = false;
      if (!first_iteration) {
        boolean_T exitg2;
        midpt = std::abs(b_x->data[0]);
        k = 0;
        exitg2 = false;
        while ((!exitg2) && (k <= b_x->size[1] - 2)) {
          tau = midpt;
          midpt = std::abs(b_x->data[k + 1]);
          if (std::abs(b_x->data[k + 1] - b_x->data[k]) <=
              2.2204460492503131E-14 * std::fmax(tau, midpt)) {
            first_iteration = true;
            exitg2 = true;
          } else {
            k++;
          }
        }
        if (first_iteration) {
          int16_T dv_idx_1;
          dv_idx_1 = static_cast<int16_T>(x->size[1]);
          ix = fx->size[0] * fx->size[1];
          fx->size[0] = 1;
          fx->size[1] = x->size[1];
          emxEnsureCapacity_real_T(fx, ix, &ic_emlrtRTEI);
          for (ix = 0; ix < dv_idx_1; ix++) {
            fx->data[ix] = 0.0;
          }
        } else {
          guard1 = true;
        }
      } else {
        guard1 = true;
      }
      if (guard1) {
        first_iteration = false;
        ix = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = b_x->size[1];
        emxEnsureCapacity_real_T(y, ix, &bb_emlrtRTEI);
        ix = b_x->size[1];
        for (k = 0; k < ix; k++) {
          y->data[k] = rt_powd_snf(b_x->data[k], fun_workspace_be - 1.0);
        }
        midpt = (fun_workspace_be + fun_workspace_ga) - 1.0;
        ix = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = b_x->size[1];
        emxEnsureCapacity_real_T(b_y, ix, &bb_emlrtRTEI);
        ix = b_x->size[1];
        for (k = 0; k < ix; k++) {
          b_y->data[k] = rt_powd_snf(b_x->data[k], midpt);
        }
        if (y->size[1] == b_y->size[1]) {
          ix = a->size[0] * a->size[1];
          a->size[0] = 1;
          a->size[1] = y->size[1];
          emxEnsureCapacity_real_T(a, ix, &pc_emlrtRTEI);
          for (ix = 0; ix < y->size[1]; ix++) {
            a->data[ix] = fun_workspace_be * y->data[ix] -
                          fun_workspace_ga * b_y->data[ix];
          }
        } else {
          binary_expand_op_5(a, fun_workspace_be, y, fun_workspace_ga, b_y);
        }
        ix = b_dv1->size[0] * b_dv1->size[1];
        b_dv1->size[0] = 1;
        b_dv1->size[1] = a->size[1];
        emxEnsureCapacity_real_T(b_dv1, ix, &bb_emlrtRTEI);
        ix = a->size[1];
        for (k = 0; k < ix; k++) {
          d = a->data[k];
          b_dv1->data[k] = d * d;
        }
        ix = c_y->size[0] * c_y->size[1];
        c_y->size[0] = 1;
        c_y->size[1] = b_x->size[1];
        emxEnsureCapacity_real_T(c_y, ix, &bb_emlrtRTEI);
        ix = b_x->size[1];
        for (k = 0; k < ix; k++) {
          c_y->data[k] = rt_powd_snf(b_x->data[k], fun_workspace_ga);
        }
        ix = dv4->size[0] * dv4->size[1];
        dv4->size[0] = 1;
        dv4->size[1] = c_y->size[1];
        emxEnsureCapacity_real_T(dv4, ix, &qc_emlrtRTEI);
        for (ix = 0; ix < c_y->size[1]; ix++) {
          dv4->data[ix] = -2.0 * c_y->data[ix];
        }
        ix = b_x->size[1];
        for (k = 0; k < ix; k++) {
          dv4->data[k] = std::exp(dv4->data[k]);
        }
        if (b_dv1->size[1] == dv4->size[1]) {
          ix = fx->size[0] * fx->size[1];
          fx->size[0] = 1;
          fx->size[1] = b_dv1->size[1];
          emxEnsureCapacity_real_T(fx, ix, &jc_emlrtRTEI);
          for (ix = 0; ix < b_dv1->size[1]; ix++) {
            fx->data[ix] = b_dv1->data[ix] * dv4->data[ix] * xt->data[ix];
          }
        } else {
          binary_expand_op_3(fx, b_dv1, dv4, xt);
        }
      }
      if (first_iteration) {
        exitg1 = 1;
      } else {
        real_T tol;
        midpt = 0.0;
        ix = -1;
        for (k = 0; k <= nsubs; k++) {
          d = 0.0;
          tau = 0.0;
          for (int32_T j{0}; j < 15; j++) {
            d += dv1[j] * fx->data[(ix + j) + 1];
            tau += dv2[j] * fx->data[(ix + j) + 1];
          }
          ix += 15;
          halfh = (subs[(k << 1) + 1] - subs[k << 1]) / 2.0;
          d *= halfh;
          qsub[k] = d;
          midpt += d;
          errsub[k] = tau * halfh;
        }
        q = midpt + q_ok;
        tol = std::fmax(1.0E-10, 1.0E-6 * std::abs(q));
        tau = 2.0 * tol / pathlen;
        midpt = 0.0;
        ix = 0;
        for (k = 0; k <= nsubs; k++) {
          d = errsub[k];
          halfh = std::abs(d);
          if (halfh <= tau * ((subs[(k << 1) + 1] - subs[k << 1]) / 2.0)) {
            err_ok += d;
            q_ok += qsub[k];
          } else {
            midpt += halfh;
            ix++;
            subs[(ix - 1) << 1] = subs[k << 1];
            subs[((ix - 1) << 1) + 1] = subs[(k << 1) + 1];
          }
        }
        midpt += std::abs(err_ok);
        if ((!std::isinf(q)) && (!std::isnan(q)) &&
            ((!std::isinf(midpt)) && (!std::isnan(midpt))) && (ix != 0) &&
            (!(midpt <= tol))) {
          nsubs = (ix << 1) - 1;
          if (nsubs + 1 > 650) {
            exitg1 = 1;
          } else {
            for (k = 0; k < ix; k++) {
              subs[((((ix - k) << 1) - 1) << 1) + 1] =
                  subs[(((ix - k) - 1) << 1) + 1];
              subs[(((ix - k) << 1) - 1) << 1] =
                  (subs[((ix - k) - 1) << 1] +
                   subs[(((ix - k) - 1) << 1) + 1]) /
                  2.0;
              subs[((((ix - k) << 1) - 2) << 1) + 1] =
                  subs[(((ix - k) << 1) - 1) << 1];
              subs[(((ix - k) << 1) - 2) << 1] = subs[((ix - k) - 1) << 1];
            }
          }
        } else {
          exitg1 = 1;
        }
      }
    } while (exitg1 == 0);
    emxFree_real_T(&dv4);
    emxFree_real_T(&c_y);
    emxFree_real_T(&b_dv1);
    emxFree_real_T(&a);
    emxFree_real_T(&b_y);
    emxFree_real_T(&fx);
    emxFree_real_T(&y);
    emxFree_real_T(&xt);
    emxFree_real_T(&b_x);
    emxFree_real_T(&x);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  return q;
}

//
//
int32_T split(real_T x[650], int32_T nx, real_T *pathlen)
{
  int32_T n_idx_0;
  int32_T nxnew;
  *pathlen = x[1] - x[0];
  if (*pathlen > 0.0) {
    real_T udelta;
    udelta = 10.0 / *pathlen;
    n_idx_0 =
        static_cast<int32_T>(std::ceil(std::abs(x[1] - x[0]) * udelta) - 1.0);
    nxnew =
        static_cast<int32_T>(std::ceil(std::abs(x[1] - x[0]) * udelta) - 1.0) +
        2;
    if (nxnew > 2) {
      x[nxnew - 1] = x[1];
      udelta = (x[1] - x[0]) / static_cast<real_T>(n_idx_0 + 1);
      for (int32_T j{0}; j < n_idx_0; j++) {
        x[(nxnew - j) - 2] = x[0] + static_cast<real_T>(n_idx_0 - j) * udelta;
      }
    }
    nx = nxnew;
  } else {
    nxnew = 2;
  }
  n_idx_0 = 0;
  for (int32_T j{0}; j <= nx - 2; j++) {
    if (std::abs(x[j + 1] - x[n_idx_0]) > 0.0) {
      n_idx_0++;
      x[n_idx_0] = x[j + 1];
    } else {
      nxnew--;
    }
  }
  if (nxnew < 2) {
    x[1] = x[nx - 1];
    nxnew = 2;
  }
  return nxnew;
}

} // namespace coder

// End of code generation (quadgk.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// vvarstd.cu
//
// Code generation for function 'vvarstd'
//

// Include files
#include "vvarstd.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
real_T vvarstd(const emxArray_real_T *v, int32_T n)
{
  real_T s;
  nvtxRangePushA("#fcn#vvarstd#" MW_AT_LOCATION);
  if (n == 0) {
    s = rtNaN;
  } else if (n == 1) {
    if ((!std::isinf(v->data[0])) && (!std::isnan(v->data[0]))) {
      s = 0.0;
    } else {
      s = rtNaN;
    }
  } else {
    real_T xbar;
    if (v->size[0] == 0) {
      xbar = 0.0;
    } else {
      xbar = v->data[0];
      profileLoopStart("vvarstd_loop_0", __LINE__, (n - 2) + 1, "");
      for (int32_T k{0}; k <= n - 2; k++) {
        xbar += v->data[k + 1];
      }
      profileLoopEnd();
    }
    xbar /= static_cast<real_T>(n);
    s = 0.0;
    profileLoopStart("vvarstd_loop_1", __LINE__, (n - 1) + 1, "");
    for (int32_T k{0}; k < n; k++) {
      real_T t;
      t = v->data[k] - xbar;
      s += t * t;
    }
    profileLoopEnd();
    s /= static_cast<real_T>(n);
  }
  nvtxRangePop();
  return s;
}

} // namespace coder

// End of code generation (vvarstd.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wt.cu
//
// Code generation for function 'wt'
//

// Include files
#include "wt.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "vvarstd.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hipfft/hipfft.h"
#include "nvtx3/nvToolsExt.h"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo
    ad_emlrtRTEI{
        105,      // lineNo
        20,       // colNo
        "varstd", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
        "varstd.m" // pName
    };

static emlrtRTEInfo bd_emlrtRTEI{
    135,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo cd_emlrtRTEI{
    143,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo dd_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "cuFFTNDCallback", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "cuFFTNDCallback.p" // pName
};

static emlrtRTEInfo ed_emlrtRTEI{
    138,  // lineNo
    19,   // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo fd_emlrtRTEI{
    138,  // lineNo
    6,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo gd_emlrtRTEI{
    161,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo hd_emlrtRTEI{
    180,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo id_emlrtRTEI{
    151,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

// Function Declarations
static __global__ void
cwtfilterbank_wt_kernel26(const int32_T i, const int32_T b, emxArray_real_T x);

static __global__ void cwtfilterbank_wt_kernel27(const uint32_T xSize_dim1,
                                                 const int32_T b,
                                                 emxArray_creal_T xposdft);

static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel26(
    const int32_T i, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    real_T xtmp;
    int32_T b_j1;
    int32_T j2;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

static __global__ __launch_bounds__(1024, 1) void cwtfilterbank_wt_kernel27(
    const uint32_T xSize_dim1, const int32_T b, emxArray_creal_T xposdft)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    b_j1 = static_cast<int32_T>(idx);
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].re =
        xposdft.data[b_j1 + 1].re;
    xposdft.data[(static_cast<int32_T>(xSize_dim1) - b_j1) - 1].im =
        -xposdft.data[b_j1 + 1].im;
  }
}

static void gpuEmxMemcpyGpuToCpu_creal_T(emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyGpuToCpu_creal_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(creal_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

//
//
namespace coder {
void cwtfilterbank_wt(cwtfilterbank *self, const emxArray_real_T *x,
                      emxArray_creal_T *varargout_1)
{
  dim3 block;
  dim3 grid;
  emxArray_creal_T gpu_cfsposdft;
  emxArray_creal_T gpu_xposdft;
  emxArray_creal_T *cpu_cfsposdft;
  emxArray_creal_T *cpu_xposdft;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xv;
  emxArray_real_T *b_x;
  emxArray_real_T *cpu_x;
  emxArray_real_T *cpu_xv;
  real_T ai;
  real_T d;
  int32_T acoef;
  int32_T bcoef;
  int32_T csz_idx_1;
  int32_T i;
  int32_T i2;
  int32_T i3;
  int32_T nd2;
  uint32_T xSize[2];
  boolean_T validLaunchParams;
  boolean_T x_outdatedOnCpu;
  boolean_T xv_outdatedOnGpu;
  nvtxRangePushA("#fcn#cwtfilterbank_wt#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfsposdft);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xv);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nd2 = x->size[0];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_x, 1, &ad_emlrtRTEI, true);
  bcoef = b_x->size[0];
  b_x->size[0] = x->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_x, bcoef, &ad_emlrtRTEI);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xv, 2, &bd_emlrtRTEI, true);
  xv_outdatedOnGpu = false;
  bcoef = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = x->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_xv, bcoef, &bd_emlrtRTEI);
  profileLoopStart("cwtfilterbank_wt_loop_0", __LINE__, (nd2 - 1) + 1, "");
  for (bcoef = 0; bcoef < nd2; bcoef++) {
    b_x->data[bcoef] = x->data[bcoef];
    cpu_xv->data[bcoef] = x->data[bcoef];
    xv_outdatedOnGpu = true;
  }
  profileLoopEnd();
  nvtxMarkA("#vvarstd#" MW_AT_LINE);
  self->sigvar = vvarstd(b_x, x->size[0]);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_x);
  if (self->SignalPad > 0.0) {
    d = self->SignalPad;
    if (d < 1.0) {
      i = 0;
    } else {
      i = static_cast<int32_T>(d);
    }
    d = (static_cast<real_T>(x->size[0]) - self->SignalPad) + 1.0;
    if (d > x->size[0]) {
      csz_idx_1 = 0;
      i2 = 1;
      i3 = -1;
    } else {
      csz_idx_1 = x->size[0] - 1;
      i2 = -1;
      i3 = static_cast<int32_T>(d) - 1;
    }
    nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
    emxInit_real_T(&cpu_x, 2, &ed_emlrtRTEI, true);
    xv_outdatedOnGpu = false;
    bcoef = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = 1;
    cpu_x->size[1] = i;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_x, bcoef, &ed_emlrtRTEI);
    profileLoopStart("cwtfilterbank_wt_loop_2", __LINE__, (i - 1) + 1, "");
    for (bcoef = 0; bcoef < i; bcoef++) {
      cpu_x->data[bcoef] = x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters((i >> 1) - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !xv_outdatedOnGpu);
    if (xv_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwtfilterbank_wt_kernel26#" MW_AT_LINE);
      cwtfilterbank_wt_kernel26<<<grid, block>>>(i, (i >> 1) - 1, gpu_x);
    }
    x_outdatedOnCpu = true;
    xv_outdatedOnGpu = false;
    bcoef = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    cpu_xv->size[1] =
        ((x->size[0] + cpu_x->size[1]) + div_s32(i3 - csz_idx_1, i2)) + 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_xv, bcoef, &fd_emlrtRTEI);
    acoef = cpu_x->size[1];
    profileLoopStart("cwtfilterbank_wt_loop_4", __LINE__, (acoef - 1) + 1, "");
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      if (x_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_x, &gpu_x);
      }
      x_outdatedOnCpu = false;
      cpu_xv->data[bcoef] = cpu_x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    acoef = x->size[0];
    profileLoopStart("cwtfilterbank_wt_loop_6", __LINE__, (acoef - 1) + 1, "");
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      cpu_xv->data[bcoef + cpu_x->size[1]] = x->data[bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    acoef = div_s32(i3 - csz_idx_1, i2);
    profileLoopStart("cwtfilterbank_wt_loop_7", __LINE__, acoef + 1, "");
    for (bcoef = 0; bcoef <= acoef; bcoef++) {
      cpu_xv->data[(bcoef + x->size[0]) + cpu_x->size[1]] =
          x->data[csz_idx_1 + i2 * bcoef];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
    emxFree_real_T(&cpu_x);
  }
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_xposdft, 2, &cd_emlrtRTEI, true);
  if (cpu_xv->size[1] == 0) {
    xv_outdatedOnGpu = false;
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = 0;
  } else if (cpu_xv->size[1] - (static_cast<int32_T>(
                                    static_cast<uint32_T>(cpu_xv->size[1]) >> 1)
                                << 1) ==
             1) {
    hipfftHandle b_fftPlanHandle;
    xv_outdatedOnGpu = false;
    bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = cpu_xv->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_xposdft, bcoef, &cd_emlrtRTEI);
    profileLoopStart("cwtfilterbank_wt_loop_1", __LINE__,
                     (cpu_xv->size[1] - 1) + 1, "");
    for (bcoef = 0; bcoef < cpu_xv->size[1]; bcoef++) {
      cpu_xposdft->data[bcoef].re = cpu_xv->data[bcoef];
      cpu_xposdft->data[bcoef].im = 0.0;
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nd2 = cpu_xposdft->size[1];
    b_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_Z2Z, 1);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_xposdft, &gpu_xposdft, !xv_outdatedOnGpu);
    if (xv_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_xposdft, cpu_xposdft);
    }
    hipfftExecZ2Z(b_fftPlanHandle, (hipfftDoubleComplex *)&gpu_xposdft.data[0],
                 (hipfftDoubleComplex *)&gpu_xposdft.data[0], HIPFFT_FORWARD);
    xv_outdatedOnGpu = true;
  } else {
    hipfftHandle fftPlanHandle;
    nd2 = cpu_xv->size[1];
    xSize[1] = static_cast<uint32_T>(cpu_xv->size[1]);
    bcoef = cpu_xposdft->size[0] * cpu_xposdft->size[1];
    cpu_xposdft->size[0] = 1;
    cpu_xposdft->size[1] = cpu_xv->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_xposdft, bcoef, &dd_emlrtRTEI);
    fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, 1, 1, HIPFFT_D2Z, 1);
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_xv, &gpu_xv, !xv_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_xposdft, &gpu_xposdft, true);
    if (xv_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_xv, cpu_xv);
    }
    hipfftExecD2Z(fftPlanHandle, (hipfftDoubleReal *)&gpu_xv.data[0],
                 (hipfftDoubleComplex *)&gpu_xposdft.data[0]);
    nd2 = static_cast<int32_T>(static_cast<real_T>(cpu_xv->size[1] + 1) / 2.0);
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(nd2 - 2), &grid, &block,
                            2147483647U);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#cwtfilterbank_wt_kernel27#" MW_AT_LINE);
      cwtfilterbank_wt_kernel27<<<grid, block>>>(xSize[1], nd2 - 2,
                                                 gpu_xposdft);
    }
    xv_outdatedOnGpu = true;
  }
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xv);
  nd2 = self->PsiDFT->size[1];
  i = cpu_xposdft->size[1];
  if (nd2 <= i) {
    i = nd2;
  }
  if (self->PsiDFT->size[1] == 1) {
    csz_idx_1 = cpu_xposdft->size[1];
  } else if (cpu_xposdft->size[1] == 1) {
    csz_idx_1 = self->PsiDFT->size[1];
  } else if (cpu_xposdft->size[1] == self->PsiDFT->size[1]) {
    csz_idx_1 = cpu_xposdft->size[1];
  } else {
    csz_idx_1 = i;
  }
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfsposdft, 2, &id_emlrtRTEI, true);
  x_outdatedOnCpu = false;
  bcoef = cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
  cpu_cfsposdft->size[0] = self->PsiDFT->size[0];
  nd2 = self->PsiDFT->size[1];
  i = cpu_xposdft->size[1];
  if (nd2 <= i) {
    i = nd2;
  }
  if (self->PsiDFT->size[1] == 1) {
    cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
  } else if (cpu_xposdft->size[1] == 1) {
    cpu_cfsposdft->size[1] = self->PsiDFT->size[1];
  } else if (cpu_xposdft->size[1] == self->PsiDFT->size[1]) {
    cpu_cfsposdft->size[1] = cpu_xposdft->size[1];
  } else {
    cpu_cfsposdft->size[1] = i;
  }
  nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
  emxEnsureCapacity_creal_T(cpu_cfsposdft, bcoef, &l_emlrtRTEI);
  if ((self->PsiDFT->size[0] != 0) && (csz_idx_1 != 0)) {
    acoef = (cpu_xposdft->size[1] != 1);
    bcoef = (self->PsiDFT->size[1] != 1);
    csz_idx_1--;
    nd2 = (self->PsiDFT->size[0] != 1);
    profileLoopStart("cwtfilterbank_wt_loop_3", __LINE__, csz_idx_1 + 1, "");
    for (int32_T k{0}; k <= csz_idx_1; k++) {
      i = acoef * k;
      i2 = bcoef * k;
      i3 = cpu_cfsposdft->size[0] - 1;
      profileLoopStart("cwtfilterbank_wt_loop_5", __LINE__, i3 + 1, "");
      for (int32_T b_k{0}; b_k <= i3; b_k++) {
        if (xv_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal_T(cpu_xposdft, &gpu_xposdft);
        }
        cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].re =
            self->PsiDFT->data[nd2 * b_k + self->PsiDFT->size[0] * i2] *
            cpu_xposdft->data[i].re;
        xv_outdatedOnGpu = false;
        cpu_cfsposdft->data[b_k + cpu_cfsposdft->size[0] * k].im =
            self->PsiDFT->data[nd2 * b_k + self->PsiDFT->size[0] * i2] *
            cpu_xposdft->data[i].im;
        x_outdatedOnCpu = true;
      }
      profileLoopEnd();
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_xposdft);
  if ((cpu_cfsposdft->size[0] == 0) || (cpu_cfsposdft->size[1] == 0)) {
    xSize[0] = static_cast<uint32_T>(cpu_cfsposdft->size[0]);
    xSize[1] = static_cast<uint32_T>(cpu_cfsposdft->size[1]);
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[0] = cpu_cfsposdft->size[0];
    varargout_1->size[1] = cpu_cfsposdft->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(varargout_1, bcoef, &gd_emlrtRTEI);
    if (static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1 >=
        0) {
      std::memset(&varargout_1->data[0], 0,
                  static_cast<uint32_T>(static_cast<int32_T>(xSize[0]) *
                                        static_cast<int32_T>(xSize[1])) *
                      sizeof(creal_T));
    }
  } else {
    hipfftHandle c_fftPlanHandle;
    nd2 = cpu_cfsposdft->size[1];
    c_fftPlanHandle = acquireCUFFTPlan(1, &nd2, &nd2, cpu_cfsposdft->size[0], 1,
                                       HIPFFT_Z2Z, cpu_cfsposdft->size[0]);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_cfsposdft, &gpu_cfsposdft,
                                 !x_outdatedOnCpu);
    if (x_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_cfsposdft, cpu_cfsposdft);
    }
    hipfftExecZ2Z(c_fftPlanHandle, (hipfftDoubleComplex *)&gpu_cfsposdft.data[0],
                 (hipfftDoubleComplex *)&gpu_cfsposdft.data[0], HIPFFT_BACKWARD);
    xv_outdatedOnGpu = true;
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[0] = cpu_cfsposdft->size[0];
    varargout_1->size[1] = cpu_cfsposdft->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(varargout_1, bcoef, &gd_emlrtRTEI);
    nd2 = cpu_cfsposdft->size[1];
    profileLoopStart("cwtfilterbank_wt_loop_8", __LINE__,
                     (cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1] - 1) + 1,
                     "");
    for (bcoef = 0; bcoef < cpu_cfsposdft->size[0] * cpu_cfsposdft->size[1];
         bcoef++) {
      if (xv_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_creal_T(cpu_cfsposdft, &gpu_cfsposdft);
      }
      d = cpu_cfsposdft->data[bcoef].re;
      xv_outdatedOnGpu = false;
      ai = cpu_cfsposdft->data[bcoef].im;
      if (ai == 0.0) {
        varargout_1->data[bcoef].re = d / static_cast<real_T>(nd2);
        varargout_1->data[bcoef].im = 0.0;
      } else if (d == 0.0) {
        varargout_1->data[bcoef].re = 0.0;
        varargout_1->data[bcoef].im = ai / static_cast<real_T>(nd2);
      } else {
        varargout_1->data[bcoef].re = d / static_cast<real_T>(nd2);
        varargout_1->data[bcoef].im = ai / static_cast<real_T>(nd2);
      }
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfsposdft);
  if (self->SignalPad > 0.0) {
    d = self->SignalPad + 1.0;
    ai = self->SignalPad + self->SignalLength;
    if (d > ai) {
      nd2 = 0;
      i = 0;
    } else {
      nd2 = static_cast<int32_T>(d) - 1;
      i = static_cast<int32_T>(ai);
    }
    csz_idx_1 = varargout_1->size[0];
    acoef = i - nd2;
    profileLoopStart("cwtfilterbank_wt_loop_9", __LINE__, (acoef - 1) + 1, "");
    for (bcoef = 0; bcoef < acoef; bcoef++) {
      profileLoopStart("cwtfilterbank_wt_loop_10", __LINE__,
                       (csz_idx_1 - 1) + 1, "");
      for (i2 = 0; i2 < csz_idx_1; i2++) {
        varargout_1->data[i2 + csz_idx_1 * bcoef] =
            varargout_1->data[i2 + varargout_1->size[0] * (nd2 + bcoef)];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    bcoef = varargout_1->size[0] * varargout_1->size[1];
    varargout_1->size[1] = i - nd2;
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(varargout_1, bcoef, &hd_emlrtRTEI);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xv);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfsposdft);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (wt.cu)

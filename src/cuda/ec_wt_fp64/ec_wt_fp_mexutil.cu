//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp_mexutil.cu
//
// Code generation for function 'ec_wt_fp_mexutil'
//

// Include files
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"
#include <cstring>

// Function Definitions
void checkCudaError(hipError_t errCode, const char_T *file, uint32_T b_line)
{
  nvtxRangePushA("#fcn#checkCudaError#" MW_AT_LOCATION);
  if (errCode != hipSuccess) {
    nvtxMarkA("#raiseCudaError#" MW_AT_LINE);
    raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                   hipGetErrorString(errCode));
  }
  nvtxRangePop();
}

uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
  }
  numIters *= n;
  nvtxRangePop();
  return numIters;
}

uint64_T computeNumIters(int32_T ub)
{
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  numIters = 0UL;
  if (ub >= 0) {
    numIters = static_cast<uint64_T>(ub + 1);
  }
  nvtxRangePop();
  return numIters;
}

real_T d_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  nvtxRangePushA("#fcn#d_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 0U,
                          (const void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  nvtxRangePop();
  return ret;
}

real_T emlrt_marshallIn(const mxArray *a__output_of_length_,
                        const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  y = emlrt_marshallIn(emlrtAlias(a__output_of_length_), &thisId);
  emlrtDestroyArray(&a__output_of_length_);
  nvtxRangePop();
  return y;
}

real_T emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier *parentId)
{
  real_T y;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#d_emlrt_marshallIn#" MW_AT_LINE);
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  nvtxRangePop();
  return y;
}

void gpuEmxEnsureCapacity_creal_T(const emxArray_creal_T *cpu,
                                  emxArray_creal_T *gpu, boolean_T needsCopy)
{
  creal_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_creal_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_creal_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(creal_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(creal_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                 emxArray_real_T *gpu, boolean_T needsCopy)
{
  real_T *newData;
  int32_T i;
  int32_T totalSizeCpu;
  int32_T totalSizeGpu;
  nvtxRangePushA("#fcn#gpuEmxEnsureCapacity_real_T#" MW_AT_LOCATION);
  if (gpu->numDimensions == 0) {
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  }
  totalSizeCpu = 1;
  totalSizeGpu = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxEnsureCapacity_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    totalSizeGpu *= gpu->size[i];
    totalSizeCpu *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
    i++;
  }
  nvtxRangePop();
  if (gpu->allocatedSize < totalSizeCpu) {
    i = cpu->allocatedSize;
    if (i < totalSizeCpu) {
      i = totalSizeCpu;
    }
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
    checkCudaError(
        mwCudaMalloc(&newData, static_cast<uint32_T>(i) * sizeof(real_T)),
        __FILE__, __LINE__);
    needsCopy = (needsCopy && gpu->canFreeData);
    if (needsCopy) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
      checkCudaError(
          hipMemcpy(newData, gpu->data,
                     static_cast<uint32_T>(totalSizeGpu) * sizeof(real_T),
                     hipMemcpyDeviceToDevice),
          __FILE__, __LINE__);
    }
    if (gpu->canFreeData) {
      nvtxMarkA("#checkCudaError#" MW_AT_LINE);
      nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
      checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
    }
    gpu->data = newData;
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
  }
  nvtxRangePop();
}

void gpuEmxFree_creal_T(emxArray_creal_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_creal_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxFree_real_T#" MW_AT_LOCATION);
  if (gpu->data != (void *)4207599121UL) {
    nvtxMarkA("#checkCudaError#" MW_AT_LINE);
    nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
    checkCudaError(mwCudaFree(gpu->data), __FILE__, __LINE__);
  }
  emlrtFreeMex(gpu->size);
  nvtxRangePop();
}

void gpuEmxMemcpyCpuToGpu_real_T(emxArray_real_T *gpu,
                                 const emxArray_real_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyCpuToGpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(gpu->data, cpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxMemcpyGpuToCpu_real_T(emxArray_real_T *cpu, emxArray_real_T *gpu)
{
  int32_T actualSize;
  int32_T i;
  nvtxRangePushA("#fcn#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LOCATION);
  actualSize = 1;
  i = 0;
  nvtxRangePushA("#loop#gpuEmxMemcpyGpuToCpu_real_T_whileloop_0##" MW_AT_LINE);
  while (i < cpu->numDimensions) {
    actualSize *= cpu->size[i];
    i++;
  }
  nvtxRangePop();
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
  checkCudaError(hipMemcpy(cpu->data, gpu->data,
                            static_cast<uint32_T>(actualSize) * sizeof(real_T),
                            hipMemcpyDeviceToHost),
                 __FILE__, __LINE__);
  nvtxRangePop();
}

void gpuEmxReset_creal_T(emxArray_creal_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_creal_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_creal_T));
  nvtxRangePop();
}

void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  nvtxRangePushA("#fcn#gpuEmxReset_real_T#" MW_AT_LOCATION);
  std::memset(gpu, 0, sizeof(emxArray_real_T));
  nvtxRangePop();
}

void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                    const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  nvtxRangePushA("#fcn#raiseCudaError#" MW_AT_LOCATION);
  len = strlen(file);
  pn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  fn = static_cast<char_T *>(calloc(len + 1UL, 1UL));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
  nvtxRangePop();
}

// End of code generation (ec_wt_fp_mexutil.cu)

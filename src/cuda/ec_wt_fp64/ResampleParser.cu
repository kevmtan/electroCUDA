#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ResampleParser.cu
//
// Code generation for function 'ResampleParser'
//

// Include files
#include "ResampleParser.h"
#include "casyi.h"
#include "cmlri.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "firls.h"
#include "gammaln.h"
#include "rt_nonfinite.h"
#include "warning.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo re_emlrtRTEI{
    694,              // lineNo
    48,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo se_emlrtRTEI{
    816,              // lineNo
    22,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo te_emlrtRTEI{
    37,                                                    // lineNo
    20,                                                    // colNo
    "kaiser",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/kaiser.m" // pName
};

static emlrtRTEInfo ue_emlrtRTEI{
    816,              // lineNo
    63,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo ve_emlrtRTEI{
    816,              // lineNo
    17,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo we_emlrtRTEI{
    819,              // lineNo
    17,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo xe_emlrtRTEI{
    777,              // lineNo
    13,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo ye_emlrtRTEI{
    778,              // lineNo
    15,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo af_emlrtRTEI{
    778,              // lineNo
    27,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

static emlrtRTEInfo bf_emlrtRTEI{
    778,              // lineNo
    13,               // colNo
    "ResampleParser", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/ResampleParser.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace b_signal {
namespace internal {
namespace resample {
void c_ResampleParser_createFilterAn(int32_T obj_dim, real_T obj_p,
                                     real_T obj_q,
                                     const real_T obj_inputSize[2],
                                     boolean_T obj_isRowVectorInput,
                                     const emxArray_creal_T *obj_x,
                                     ResampleParser *obj)
{
  emxArray_int32_T *iv;
  emxArray_real_T *b_dv1;
  emxArray_real_T *c_dv;
  emxArray_real_T *h1;
  emxArray_real_T *r;
  emxArray_uint32_T *y;
  creal_T tmp;
  creal_T zd;
  real_T ak;
  real_T fc;
  real_T pqmax;
  int32_T inw;
  int32_T iseven;
  int32_T nw;
  nvtxRangePushA("#fcn#c_ResampleParser_createFilterAn#" MW_AT_LOCATION);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  obj->dim = obj_dim;
  obj->p = obj_p;
  obj->q = obj_q;
  obj->inputSize[0] = obj_inputSize[0];
  obj->inputSize[1] = obj_inputSize[1];
  obj->isRowVectorInput = obj_isRowVectorInput;
  inw = obj->x->size[0] * obj->x->size[1];
  obj->x->size[0] = obj_x->size[0];
  obj->x->size[1] = obj_x->size[1];
  nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
  emxEnsureCapacity_creal_T(obj->x, inw, &re_emlrtRTEI);
  profileLoopStart("c_ResampleParser_createFilterAn_loop_0", __LINE__,
                   (obj_x->size[0] * obj_x->size[1] - 1) + 1, "");
  for (inw = 0; inw < obj_x->size[0] * obj_x->size[1]; inw++) {
    obj->x->data[inw] = obj_x->data[inw];
  }
  real_T b_dv[4];
  profileLoopEnd();
  pqmax = std::fmax(obj_p, obj_q);
  fc = 0.5 / pqmax;
  pqmax = 20.0 * pqmax + 1.0;
  b_dv[0] = 0.0;
  b_dv[1] = 2.0 * fc;
  b_dv[2] = 2.0 * fc;
  b_dv[3] = 1.0;
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&r, 2, &se_emlrtRTEI, true);
  nvtxMarkA("#firls#" MW_AT_LINE);
  firls(pqmax - 1.0, b_dv, r);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_dv, 1, &se_emlrtRTEI, true);
  inw = c_dv->size[0];
  c_dv->size[0] = r->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(c_dv, inw, &se_emlrtRTEI);
  profileLoopStart("c_ResampleParser_createFilterAn_loop_1", __LINE__,
                   (r->size[1] - 1) + 1, "");
  for (inw = 0; inw < r->size[1]; inw++) {
    c_dv->data[inw] = r->data[inw];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&r);
  if (pqmax == std::floor(pqmax)) {
    nw = static_cast<int32_T>(pqmax);
  } else {
    nw = static_cast<int32_T>(std::round(pqmax));
    nvtxMarkA("#warning#" MW_AT_LINE);
    ::coder::internal::warning();
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_dv1, 1, &se_emlrtRTEI, true);
  inw = b_dv1->size[0];
  b_dv1->size[0] = nw;
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(b_dv1, inw, &te_emlrtRTEI);
  if (nw <= 1) {
    inw = b_dv1->size[0];
    b_dv1->size[0] = nw;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(b_dv1, inw, &ue_emlrtRTEI);
    profileLoopStart("c_ResampleParser_createFilterAn_loop_2", __LINE__,
                     (nw - 1) + 1, "");
    for (inw = 0; inw < nw; inw++) {
      b_dv1->data[inw] = 1.0;
    }
    profileLoopEnd();
  } else {
    int64_T b;
    int32_T mid;
    iseven = 1 - (nw & 1);
    mid = (nw >> 1) + 1;
    nvtxMarkA("#computeEndIdx#" MW_AT_LINE);
    if (computeEndIdx(static_cast<int64_T>(mid), static_cast<int64_T>(nw),
                      1L) >= 0L) {
      zd.im = 0.0;
    }
    nvtxMarkA("#computeEndIdx#" MW_AT_LINE);
    b = computeEndIdx(static_cast<int64_T>(mid), static_cast<int64_T>(nw), 1L);
    profileLoopStart("c_ResampleParser_createFilterAn_loop_3", __LINE__, b + 1L,
                     "");
    for (int64_T k{0L}; k <= b; k++) {
      pqmax = static_cast<real_T>(
                  iseven + ((static_cast<int32_T>(mid + k) - mid) << 1)) /
              (static_cast<real_T>(nw) - 1.0);
      zd.re = 5.0 * std::sqrt((1.0 - pqmax) * (pqmax + 1.0));
      if (std::isnan(zd.re)) {
        tmp.re = rtNaN;
        tmp.im = 0.0;
      } else {
        real_T az;
        int32_T b_nw;
        int32_T ierr;
        boolean_T guard1;
        ierr = 0;
        nvtxMarkA("#rt_hypotd_snf#" MW_AT_LINE);
        pqmax = rt_hypotd_snf(zd.re, 0.0);
        if (pqmax > 1.0737418235E+9) {
          ierr = 4;
        } else if (pqmax > 32767.999992370605) {
          ierr = 3;
        }
        tmp.re = 0.0;
        tmp.im = 0.0;
        nvtxMarkA("#rt_hypotd_snf#" MW_AT_LINE);
        az = rt_hypotd_snf(zd.re, 0.0);
        guard1 = false;
        if (az <= 2.0) {
          b_nw = 0;
          nvtxMarkA("#rt_hypotd_snf#" MW_AT_LINE);
          pqmax = rt_hypotd_snf(zd.re, 0.0);
          if (pqmax == 0.0) {
            tmp.re = 1.0;
            tmp.im = 0.0;
          } else if (pqmax < 2.2250738585072014E-305) {
            tmp.re = 1.0;
            tmp.im = 0.0;
          } else {
            real_T acz;
            real_T cz_re;
            fc = 0.5 * zd.re;
            if (pqmax > 4.7170688552396617E-153) {
              cz_re = fc * fc;
              nvtxMarkA("#rt_hypotd_snf#" MW_AT_LINE);
              acz = rt_hypotd_snf(cz_re, 0.0);
            } else {
              cz_re = 0.0;
              acz = 0.0;
            }
            pqmax = 1.0;
            nvtxMarkA("#gammaln#" MW_AT_LINE);
            gammaln(&pqmax);
            fc = std::log(fc) * 0.0 - pqmax;
            if (fc > -700.92179369444591) {
              real_T b_atol;
              real_T coef_im;
              real_T coef_re;
              real_T s1_im;
              real_T s1_re;
              coef_re = std::exp(fc);
              coef_im = std::exp(fc) * 0.0;
              b_atol = 2.2204460492503131E-16 * acz;
              s1_re = 1.0;
              s1_im = 0.0;
              if (!(acz < 2.2204460492503131E-16)) {
                real_T aa;
                real_T s;
                fc = 1.0;
                pqmax = 0.0;
                ak = 3.0;
                s = 1.0;
                aa = 2.0;
                nvtxRangePushA("#loop#c_ResampleParser_createFilterAn_"
                               "whileloop_1##" MW_AT_LINE);
                real_T re;
                real_T rs;
                do {
                  rs = 1.0 / s;
                  re = fc * cz_re - pqmax * 0.0;
                  pqmax = fc * 0.0 + pqmax * cz_re;
                  fc = rs * re;
                  pqmax *= rs;
                  s1_re += fc;
                  s1_im += pqmax;
                  s += ak;
                  ak += 2.0;
                  aa = aa * acz * rs;
                } while (!!(aa > b_atol));
                nvtxRangePop();
              }
              pqmax = s1_re * coef_re - s1_im * coef_im;
              s1_im = s1_re * coef_im + s1_im * coef_re;
              tmp.re = pqmax - s1_im * 0.0;
              tmp.im = pqmax * 0.0 + s1_im;
            } else {
              b_nw = 1;
              if (acz > 0.0) {
                b_nw = -1;
              }
            }
          }
          if (b_nw < 0) {
            inw = 1;
          } else {
            inw = b_nw;
          }
          if ((1 - inw != 0) && (b_nw < 0)) {
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1) {
          if (az < 21.784271729432426) {
            nvtxMarkA("#cmlri#" MW_AT_LINE);
            b_nw = cmlri(zd, 0.0, 1, &tmp);
            if (b_nw < 0) {
              if (b_nw == -2) {
                inw = -2;
              } else {
                inw = -1;
              }
            } else {
              inw = 0;
            }
          } else {
            nvtxMarkA("#casyi#" MW_AT_LINE);
            b_nw = casyi(zd, 0.0, 1, &tmp);
            if (b_nw < 0) {
              if (b_nw == -2) {
                inw = -2;
              } else {
                inw = -1;
              }
            } else {
              inw = 0;
            }
          }
        }
        guard1 = false;
        if (inw < 0) {
          if (inw == -2) {
            tmp.re = rtNaN;
            tmp.im = 0.0;
          } else {
            ierr = 2;
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1 && (ierr == 2)) {
          tmp.re = rtInf;
          tmp.im = 0.0;
        }
        if (zd.re > 0.0) {
          pqmax = tmp.re;
          tmp.re = pqmax;
          tmp.im = 0.0;
        }
      }
      if (tmp.im == 0.0) {
        pqmax = tmp.re / 27.239871823604449;
        fc = 0.0;
      } else if (tmp.re == 0.0) {
        pqmax = 0.0;
        fc = tmp.im / 27.239871823604449;
      } else {
        pqmax = tmp.re / 27.239871823604449;
        fc = tmp.im / 27.239871823604449;
      }
      nvtxMarkA("#rt_hypotd_snf#" MW_AT_LINE);
      b_dv1->data[static_cast<int32_T>(mid + k) - 1] = rt_hypotd_snf(pqmax, fc);
    }
    profileLoopEnd();
    profileLoopStart("c_ResampleParser_createFilterAn_loop_4", __LINE__,
                     (mid - 2) + 1, "");
    for (iseven = 0; iseven <= mid - 2; iseven++) {
      b_dv1->data[iseven] = b_dv1->data[(nw - iseven) - 1];
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&h1, 1, &ve_emlrtRTEI, true);
  inw = h1->size[0];
  h1->size[0] = c_dv->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(h1, inw, &ve_emlrtRTEI);
  profileLoopStart("c_ResampleParser_createFilterAn_loop_5", __LINE__,
                   (c_dv->size[0] - 1) + 1, "");
  for (inw = 0; inw < c_dv->size[0]; inw++) {
    h1->data[inw] = c_dv->data[inw] * b_dv1->data[inw];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_dv1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_dv);
  inw = h1->size[0];
  if (h1->size[0] == 0) {
    pqmax = 0.0;
  } else {
    pqmax = h1->data[0];
    profileLoopStart("c_ResampleParser_createFilterAn_loop_6", __LINE__,
                     (inw - 2) + 1, "");
    for (iseven = 0; iseven <= inw - 2; iseven++) {
      pqmax += h1->data[iseven + 1];
    }
    profileLoopEnd();
  }
  inw = obj->filter->size[0];
  obj->filter->size[0] = h1->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(obj->filter, inw, &we_emlrtRTEI);
  profileLoopStart("c_ResampleParser_createFilterAn_loop_7", __LINE__,
                   (h1->size[0] - 1) + 1, "");
  for (inw = 0; inw < h1->size[0]; inw++) {
    obj->filter->data[inw] = obj_p * h1->data[inw] / pqmax;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&h1);
  fc = (static_cast<real_T>(obj->filter->size[0]) - 1.0) / 2.0;
  pqmax = fc;
  if (obj_q == 0.0) {
    if (fc == 0.0) {
      pqmax = 0.0;
    }
  } else if (std::isnan(obj_q)) {
    pqmax = rtNaN;
  } else if (fc == 0.0) {
    pqmax = 0.0 / obj_q;
  } else if (std::isinf(obj_q)) {
    if (fc < 0.0) {
      pqmax = obj_q;
    }
  } else {
    pqmax = std::fmod(fc, obj_q);
    if (pqmax == 0.0) {
      pqmax = obj_q * 0.0;
    } else if (fc < 0.0) {
      pqmax += obj_q;
    }
  }
  ak = std::floor(obj_q - pqmax);
  obj->filterDelay = std::floor(std::ceil(fc + ak) / obj_q);
  pqmax = static_cast<real_T>(obj->filter->size[0]) + ak;
  fc = 0.0;
  inw = obj_x->size[0];
  nvtxRangePushA(
      "#loop#c_ResampleParser_createFilterAn_whileloop_0##" MW_AT_LINE);
  while (std::ceil((((static_cast<real_T>(inw) - 1.0) * obj_p + pqmax) + fc) /
                   obj_q) -
             obj->filterDelay <
         std::ceil(static_cast<real_T>(inw) * obj_p / obj_q)) {
    fc++;
  }
  nvtxRangePop();
  inw = obj->filterWithPadding->size[0];
  obj->filterWithPadding->size[0] = static_cast<int32_T>(
      (ak + static_cast<real_T>(obj->filter->size[0])) + fc);
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(obj->filterWithPadding, inw, &xe_emlrtRTEI);
  profileLoopStart("c_ResampleParser_createFilterAn_loop_8", __LINE__,
                   (static_cast<int32_T>(
                        (ak + static_cast<real_T>(obj->filter->size[0])) + fc) -
                    1) +
                       1,
                   "");
  for (inw = 0;
       inw < static_cast<int32_T>(
                 (ak + static_cast<real_T>(obj->filter->size[0])) + fc);
       inw++) {
    obj->filterWithPadding->data[inw] = 0.0;
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_uint32_T#" MW_AT_LINE);
  emxInit_uint32_T(&y, 2, &af_emlrtRTEI, true);
  if (obj->filter->size[0] < 1) {
    y->size[0] = 1;
    y->size[1] = 0;
  } else {
    uint32_T u;
    u = static_cast<uint32_T>(obj->filter->size[0]);
    inw = y->size[0] * y->size[1];
    y->size[0] = 1;
    y->size[1] = obj->filter->size[0];
    nvtxMarkA("#emxEnsureCapacity_uint32_T#" MW_AT_LINE);
    emxEnsureCapacity_uint32_T(y, inw, &ib_emlrtRTEI);
    profileLoopStart("c_ResampleParser_createFilterAn_loop_9", __LINE__,
                     (static_cast<int32_T>(u) - 1) + 1, "");
    for (inw = 0; inw < static_cast<int32_T>(u); inw++) {
      y->data[inw] = static_cast<uint32_T>(inw) + 1U;
    }
    profileLoopEnd();
  }
  nvtxMarkA("#emxInit_int32_T#" MW_AT_LINE);
  emxInit_int32_T(&iv, 2, &bf_emlrtRTEI, true);
  inw = iv->size[0] * iv->size[1];
  iv->size[0] = 1;
  iv->size[1] = y->size[1];
  nvtxMarkA("#emxEnsureCapacity_int32_T#" MW_AT_LINE);
  emxEnsureCapacity_int32_T(iv, inw, &ye_emlrtRTEI);
  profileLoopStart("c_ResampleParser_createFilterAn_loop_10", __LINE__,
                   (y->size[1] - 1) + 1, "");
  for (inw = 0; inw < y->size[1]; inw++) {
    iv->data[inw] =
        static_cast<int32_T>(ak + static_cast<real_T>(y->data[inw]));
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_uint32_T#" MW_AT_LINE);
  emxFree_uint32_T(&y);
  profileLoopStart("c_ResampleParser_createFilterAn_loop_11", __LINE__,
                   (iv->size[1] - 1) + 1, "");
  for (inw = 0; inw < iv->size[1]; inw++) {
    obj->filterWithPadding->data[iv->data[inw] - 1] = obj->filter->data[inw];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_int32_T#" MW_AT_LINE);
  emxFree_int32_T(&iv);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxRangePop();
}

} // namespace resample
} // namespace internal
} // namespace b_signal
} // namespace coder

// End of code generation (ResampleParser.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp_initialize.cu
//
// Code generation for function 'ec_wt_fp_initialize'
//

// Include files
#include "ec_wt_fp_initialize.h"
#include "_coder_ec_wt_fp_mex.h"
#include "cwt.h"
#include "ec_wt_fp_data.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"

// Function Declarations
static void ec_wt_fp_once();

// Function Definitions
static void ec_wt_fp_once()
{
  mex_InitInfAndNan();
  nvtxRangePushA("#initFcn#ec_wt_fp_once#" MW_AT_LOCATION);
  // Initialize GPU by calling hipFree(nullptr)
  nvtxMarkA("#hipFree#" MW_AT_LINE);
  hipFree(nullptr);
  nvtxMarkA("#cwt_emx_init#" MW_AT_LINE);
  cwt_emx_init();
  nvtxMarkA("#cwt_init#" MW_AT_LINE);
  cwt_init();
  nvtxRangePop();
}

void ec_wt_fp_initialize()
{
  nvtxRangePushA("#fcn#ec_wt_fp_initialize#" MW_AT_LOCATION);
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  nvtxMarkA("#mexFunctionCreateRootTLS#" MW_AT_LINE);
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "wavelet_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "signal_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    nvtxMarkA("#ec_wt_fp_once#" MW_AT_LINE);
    ec_wt_fp_once();
  }
  nvtxRangePop();
}

// End of code generation (ec_wt_fp_initialize.cu)

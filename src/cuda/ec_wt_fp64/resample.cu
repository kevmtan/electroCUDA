#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// resample.cu
//
// Code generation for function 'resample'
//

// Include files
#include "resample.h"
#include "ResampleParser.h"
#include "conv2.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "uniformResampleKernel.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hip/hip_math_constants.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo ge_emlrtRTEI{
    154,                                                        // lineNo
    1,                                                          // colNo
    "resample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/resample.m" // pName
};

static emlrtRTEInfo he_emlrtRTEI{
    38,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo ie_emlrtRTEI{
    161,                                                        // lineNo
    9,                                                          // colNo
    "resample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/resample.m" // pName
};

static emlrtRTEInfo je_emlrtRTEI{
    21,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo ke_emlrtRTEI{
    31,                      // lineNo
    29,                      // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo le_emlrtRTEI{
    20,               // lineNo
    9,                // colNo
    "reorientOutput", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/reorientOutput.m" // pName
};

// Function Declarations
static __global__ void resample_kernel24(const emxArray_creal_T varargin_1,
                                         const int32_T b,
                                         emxArray_creal_T opts_x);

static __global__ void resample_kernel25(const emxArray_creal_T w,
                                         const int32_T u0, const int32_T b,
                                         emxArray_creal_T y);

static __global__ void resample_kernel26(emxArray_real_T y);

static __global__ void
resample_kernel27(const emxArray_creal_T y, const emxArray_real_T b_y,
                  const real_T nlast, const int32_T exponent, const int32_T b,
                  emxArray_creal_T varargout_1, int32_T varargout_1_dim0);

static __global__ void resample_kernel28(const int32_T b, emxArray_real_T y);

static __global__ void resample_kernel29(const emxArray_creal_T varargout_1,
                                         const int32_T b_varargout_1,
                                         const int32_T c_varargout_1,
                                         emxArray_creal_T d_varargout_1,
                                         int32_T varargout_1_dim0,
                                         int32_T b_varargout_1_dim0);

static __global__ void resample_kernel30(const emxArray_creal_T varargout_1,
                                         const int32_T b_varargout_1,
                                         emxArray_creal_T c_varargout_1);

static __global__ void resample_kernel31(const emxArray_creal_T varargin_1,
                                         const int32_T b_varargin_1,
                                         emxArray_creal_T opts_x);

static __global__ void
resample_kernel32(const emxArray_creal_T varargin_1, const int32_T b_varargin_1,
                  const int32_T c_varargin_1, emxArray_creal_T opts_x,
                  int32_T opts_x_dim0, int32_T varargin_1_dim0);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void resample_kernel24(
    const emxArray_creal_T varargin_1, const int32_T b, emxArray_creal_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = varargin_1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel25(
    const emxArray_creal_T w, const int32_T u0, const int32_T b,
    emxArray_creal_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = w.data[u0 * i1];
  }
}

static __global__ __launch_bounds__(32,
                                    1) void resample_kernel26(emxArray_real_T y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    y.data[0] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel27(
    const emxArray_creal_T y, const emxArray_real_T b_y, const real_T nlast,
    const int32_T exponent, const int32_T b, emxArray_creal_T varargout_1,
    int32_T varargout_1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    varargout_1.data[i1 + varargout_1_dim0 * exponent] =
        y.data[static_cast<int32_T>(nlast + b_y.data[i1]) - 1];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void resample_kernel28(const int32_T b,
                                                              emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<real_T>(i1) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel29(
    const emxArray_creal_T varargout_1, const int32_T b_varargout_1,
    const int32_T c_varargout_1, emxArray_creal_T d_varargout_1,
    int32_T varargout_1_dim0, int32_T b_varargout_1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_varargout_1) + 1UL) *
                (static_cast<uint64_T>(b_varargout_1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx %
                              (static_cast<uint64_T>(b_varargout_1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b_varargout_1) + 1UL));
    d_varargout_1.data[i2 + varargout_1_dim0 * i1] =
        varargout_1.data[i1 + b_varargout_1_dim0 * i2];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel30(
    const emxArray_creal_T varargout_1, const int32_T b_varargout_1,
    emxArray_creal_T c_varargout_1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_varargout_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_varargout_1.data[i1] = varargout_1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel31(
    const emxArray_creal_T varargin_1, const int32_T b_varargin_1,
    emxArray_creal_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_varargin_1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = varargin_1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void resample_kernel32(
    const emxArray_creal_T varargin_1, const int32_T b_varargin_1,
    const int32_T c_varargin_1, emxArray_creal_T opts_x, int32_T opts_x_dim0,
    int32_T varargin_1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_varargin_1) + 1UL) *
                (static_cast<uint64_T>(b_varargin_1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 =
        static_cast<int32_T>(idx % (static_cast<uint64_T>(b_varargin_1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b_varargin_1) + 1UL));
    opts_x.data[i2 + opts_x_dim0 * i1] =
        varargin_1.data[i1 + varargin_1_dim0 * i2];
  }
}

//
//
namespace coder {
void resample(emxArray_creal_T *cpu_varargin_1,
              emxArray_creal_T *gpu_varargin_1,
              boolean_T *varargin_1_outdatedOnGpu, real_T varargin_2,
              real_T varargin_3, emxArray_creal_T *cpu_varargout_1,
              boolean_T *varargout_1_outdatedOnCpu,
              emxArray_creal_T *gpu_varargout_1,
              boolean_T *varargout_1_outdatedOnGpu)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  b_signal::internal::resample::ResampleParser expl_temp;
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_varargout_1;
  emxArray_creal_T b_yCol;
  emxArray_creal_T gpu_opts_x;
  emxArray_creal_T gpu_w;
  emxArray_creal_T gpu_y;
  emxArray_creal_T *b_cpu_varargout_1;
  emxArray_creal_T *cpu_opts_x;
  emxArray_creal_T *cpu_w;
  emxArray_creal_T *cpu_y;
  emxArray_creal_T *yCol;
  emxArray_real_T b_gpu_y;
  emxArray_real_T *b_cpu_y;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T varargin_1[2];
  real_T Ly;
  real_T absx;
  real_T dlast;
  real_T n;
  real_T x;
  real_T xin;
  int32_T b_expl_temp[1];
  int32_T b_varargin_1;
  int32_T exponent;
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T opts_dim;
  int32_T u1;
  char_T unusedExpr[23];
  boolean_T validLaunchParams;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  nvtxRangePushA("#fcn#resample#" MW_AT_LOCATION "#M100");
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_w);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_varargout_1);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_opts_x);
  w_outdatedOnGpu = false;
  y_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_varargin_1->size[0] == 1) || (cpu_varargin_1->size[1] == 1)) {
    opts_dim = 1;
  } else {
    opts_dim = 2;
    if (cpu_varargin_1->size[0] != 1) {
      opts_dim = 1;
    }
  }
  w_outdatedOnCpu = (cpu_varargin_1->size[0] == 1);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_opts_x, 2, &ge_emlrtRTEI, true);
  if (opts_dim == 1) {
    if ((cpu_varargin_1->size[0] == 1) || (cpu_varargin_1->size[1] == 1)) {
      if ((cpu_varargin_1->size[0] == 0) || (cpu_varargin_1->size[1] == 0)) {
        u1 = 0;
      } else {
        b_varargin_1 = cpu_varargin_1->size[0];
        u1 = cpu_varargin_1->size[1];
        if (b_varargin_1 >= u1) {
          u1 = b_varargin_1;
        }
      }
      i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
      cpu_opts_x->size[0] = u1;
      cpu_opts_x->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_opts_x, i, &ge_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(u1 - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_varargin_1, gpu_varargin_1,
                                   !*varargin_1_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_opts_x, &gpu_opts_x, true);
      if (*varargin_1_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(gpu_varargin_1, cpu_varargin_1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#resample_kernel24#" MW_AT_LINE);
        resample_kernel24<<<grid, block>>>(*gpu_varargin_1, u1 - 1, gpu_opts_x);
      }
    } else {
      i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
      cpu_opts_x->size[0] = cpu_varargin_1->size[0];
      cpu_opts_x->size[1] = cpu_varargin_1->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_opts_x, i, &ge_emlrtRTEI);
      i = cpu_varargin_1->size[0] * cpu_varargin_1->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_varargin_1, gpu_varargin_1,
                                   !*varargin_1_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_opts_x, &gpu_opts_x, true);
      if (*varargin_1_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(gpu_varargin_1, cpu_varargin_1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#resample_kernel31#" MW_AT_LINE);
        resample_kernel31<<<grid, block>>>(*gpu_varargin_1, i, gpu_opts_x);
      }
    }
  } else {
    i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
    cpu_opts_x->size[0] = cpu_varargin_1->size[1];
    cpu_opts_x->size[1] = cpu_varargin_1->size[0];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(cpu_opts_x, i, &ge_emlrtRTEI);
    i = cpu_varargin_1->size[0] - 1;
    b_varargin_1 = cpu_varargin_1->size[1] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(i, b_varargin_1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_varargin_1, gpu_varargin_1,
                                 !*varargin_1_outdatedOnGpu);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal_T(cpu_opts_x, &gpu_opts_x, true);
    if (*varargin_1_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(gpu_varargin_1, cpu_varargin_1);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#resample_kernel32#" MW_AT_LINE);
      resample_kernel32<<<grid, block>>>(*gpu_varargin_1, b_varargin_1, i,
                                         gpu_opts_x, cpu_opts_x->size[0U],
                                         cpu_varargin_1->size[0U]);
    }
  }
  x = varargin_2 / varargin_3;
  xin = x;
  absx = std::abs(x);
  if (std::isinf(absx) || std::isnan(absx)) {
    absx = rtNaN;
  } else if (absx < 4.4501477170144028E-308) {
    absx = 4.94065645841247E-324;
  } else {
    std::frexp(absx, &exponent);
    absx = std::ldexp(1.0, exponent - 53);
  }
  absx = std::fmax(1.0E-12, absx);
  if (std::isinf(x) || std::isnan(x)) {
    n = !std::isnan(x);
    absx = 0.0;
  } else {
    real_T d;
    real_T nlast;
    n = 1.0;
    d = 0.0;
    nlast = 0.0;
    dlast = 1.0;
    nvtxRangePushA("#loop#resample_whileloop_0##" MW_AT_LINE);
    real_T rndx;
    int32_T exitg1;
    do {
      exitg1 = 0;
      rndx = std::round(x);
      if (!std::isinf(x)) {
        real_T tmp;
        x -= rndx;
        tmp = nlast;
        nlast = n;
        n = n * rndx + tmp;
        tmp = dlast;
        dlast = d;
        d = d * rndx + tmp;
      } else {
        nlast = n;
        dlast = d;
        n = x;
        d = 0.0;
      }
      if ((x == 0.0) || (std::abs(n / d - xin) <= absx)) {
        exitg1 = 1;
      } else {
        x = 1.0 / x;
      }
    } while (exitg1 == 0);
    nvtxRangePop();
    if (std::isnan(d)) {
      x = rtNaN;
    } else if (d < 0.0) {
      x = -1.0;
    } else {
      x = (d > 0.0);
    }
    n /= x;
    absx = std::abs(d);
  }
  profileLoopStart("resample_loop_0", __LINE__, 1 + 1, "");
  for (i = 0; i < 2; i++) {
    varargin_1[i] = cpu_varargin_1->size[i];
  }
  profileLoopEnd();
  nvtxMarkA("#emxInitStruct_ResampleParser#" MW_AT_LINE);
  emxInitStruct_ResampleParser(&expl_temp, &ge_emlrtRTEI, true);
  nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
  gpuEmxMemcpyGpuToCpu_creal_T(cpu_opts_x, &gpu_opts_x);
  nvtxMarkA("#c_ResampleParser_createFilterAn#" MW_AT_LINE);
  b_signal::internal::resample::c_ResampleParser_createFilterAn(
      opts_dim, n, absx, varargin_1, w_outdatedOnCpu, cpu_opts_x, &expl_temp);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_opts_x);
  absx = expl_temp.p;
  n = expl_temp.q;
  if (expl_temp.dim == 1) {
    if (expl_temp.isRowVectorInput) {
      *varargout_1_outdatedOnCpu = false;
      *varargout_1_outdatedOnGpu = false;
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = expl_temp.x->size[0];
      cpu_varargout_1->size[1] = expl_temp.x->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &je_emlrtRTEI);
      profileLoopStart("resample_loop_3", __LINE__,
                       (expl_temp.x->size[0] * expl_temp.x->size[1] - 1) + 1,
                       "");
      for (i = 0; i < expl_temp.x->size[0] * expl_temp.x->size[1]; i++) {
        cpu_varargout_1->data[i] = expl_temp.x->data[i];
        *varargout_1_outdatedOnGpu = true;
      }
      profileLoopEnd();
      nvtxMarkA("#uniformResampleAlongFirstDim#" MW_AT_LINE);
      b_signal::internal::resample::uniformResampleAlongFirstDim(
          cpu_varargout_1, varargout_1_outdatedOnCpu, gpu_varargout_1,
          varargout_1_outdatedOnGpu, expl_temp.p, expl_temp.q,
          expl_temp.filterWithPadding, expl_temp.filterDelay);
      if ((cpu_varargout_1->size[0] == 0) || (cpu_varargout_1->size[1] == 0)) {
        u1 = 0;
      } else {
        b_varargin_1 = cpu_varargout_1->size[0];
        u1 = cpu_varargout_1->size[1];
        if (b_varargin_1 >= u1) {
          u1 = b_varargin_1;
        }
      }
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = 1;
      cpu_varargout_1->size[1] = u1;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &ie_emlrtRTEI);
    } else {
      *varargout_1_outdatedOnCpu = false;
      *varargout_1_outdatedOnGpu = false;
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = expl_temp.x->size[0];
      cpu_varargout_1->size[1] = expl_temp.x->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &ie_emlrtRTEI);
      profileLoopStart("resample_loop_2", __LINE__,
                       (expl_temp.x->size[0] * expl_temp.x->size[1] - 1) + 1,
                       "");
      for (i = 0; i < expl_temp.x->size[0] * expl_temp.x->size[1]; i++) {
        cpu_varargout_1->data[i] = expl_temp.x->data[i];
        *varargout_1_outdatedOnGpu = true;
      }
      profileLoopEnd();
      nvtxMarkA("#uniformResampleAlongFirstDim#" MW_AT_LINE);
      b_signal::internal::resample::uniformResampleAlongFirstDim(
          cpu_varargout_1, varargout_1_outdatedOnCpu, gpu_varargout_1,
          varargout_1_outdatedOnGpu, expl_temp.p, expl_temp.q,
          expl_temp.filterWithPadding, expl_temp.filterDelay);
    }
  } else {
    boolean_T c_varargout_1_needsGpuEnsureCap;
    if (expl_temp.x->size[0] == 1) {
      dlast = std::ceil(expl_temp.p / expl_temp.q);
      *varargout_1_outdatedOnCpu = false;
      *varargout_1_outdatedOnGpu = false;
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = static_cast<int32_T>(dlast);
      cpu_varargout_1->size[1] = expl_temp.x->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &ke_emlrtRTEI);
      c_varargout_1_needsGpuEnsureCap = true;
      opts_dim = expl_temp.x->size[0] * expl_temp.x->size[1];
      if (opts_dim - 1 >= 0) {
        if ((static_cast<int32_T>(expl_temp.p) == 0) ||
            ((static_cast<int32_T>(expl_temp.p) > 0) &&
             (static_cast<int32_T>(expl_temp.p) < 1))) {
          i1 = 1;
          i2 = 0;
        } else {
          i1 = static_cast<int32_T>(expl_temp.p);
          i2 = static_cast<int32_T>(expl_temp.p);
        }
        Ly = std::ceil(
            (0.0 * expl_temp.p +
             static_cast<real_T>(expl_temp.filterWithPadding->size[0])) /
            expl_temp.q);
      }
      nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
      emxInit_creal_T(&yCol, 1, &pe_emlrtRTEI, true);
      nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
      emxInit_creal_T(&cpu_w, 1, &ne_emlrtRTEI, true);
      nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
      emxInit_creal_T(&cpu_y, 1, &ne_emlrtRTEI, true);
      nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
      emxInit_real_T(&b_cpu_y, 2, &qe_emlrtRTEI, true);
      profileLoopStart("resample_loop_4", __LINE__, (opts_dim - 1) + 1, "");
      for (exponent = 0; exponent < opts_dim; exponent++) {
        if (!(absx - 1.0 >= 0.0)) {
          y = nullptr;
          m = emlrtCreateCharArray(2, &iv[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
          emlrtAssign(&y, m);
          b_y = nullptr;
          m1 = emlrtCreateDoubleScalar(absx - 1.0);
          emlrtAssign(&b_y, m1);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(y, b_y, &f_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        i = yCol->size[0];
        yCol->size[0] = static_cast<int32_T>(absx);
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(yCol, i, &me_emlrtRTEI);
        profileLoopStart("resample_loop_5", __LINE__,
                         (static_cast<int32_T>(absx) - 1) + 1, "");
        for (i = 0; i < static_cast<int32_T>(absx); i++) {
          yCol->data[i].re = 0.0;
          yCol->data[i].im = 0.0;
        }
        profileLoopEnd();
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        b_varargin_1 = div_s32(i2 - 1, i1) + 1;
        profileLoopStart("resample_loop_6", __LINE__, (b_varargin_1 - 1) + 1,
                         "");
        for (i = 0; i < b_varargin_1; i++) {
          yCol->data[i1 * i] = expl_temp.x->data[exponent];
        }
        profileLoopEnd();
        b_yCol = *yCol;
        b_expl_temp[0] = static_cast<int32_T>(absx);
        b_yCol.size = &b_expl_temp[0];
        b_yCol.numDimensions = 1;
        nvtxMarkA("#b_conv2#" MW_AT_LINE);
        b_conv2(&b_yCol, expl_temp.filterWithPadding, cpu_w, &w_outdatedOnCpu,
                &gpu_w, &w_outdatedOnGpu);
        if (!(n - 1.0 >= 0.0)) {
          c_y = nullptr;
          m2 = emlrtCreateCharArray(2, &iv1[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
          emlrtAssign(&c_y, m2);
          d_y = nullptr;
          m3 = emlrtCreateDoubleScalar(n - 1.0);
          emlrtAssign(&d_y, m3);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(c_y, d_y, &f_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        if (n == 0.0) {
          b_varargin_1 = 1;
          u1 = -1;
        } else {
          b_varargin_1 = static_cast<int32_T>(n);
          u1 = cpu_w->size[0] - 1;
        }
        i = cpu_y->size[0];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        cpu_y->size[0] = div_s32(u1, b_varargin_1) + 1;
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_y, i, &ne_emlrtRTEI);
        i = u1 / b_varargin_1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_y, &gpu_y, true);
        w_outdatedOnCpu = false;
        if (w_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_creal_T(&gpu_w, cpu_w);
        }
        w_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#resample_kernel25#" MW_AT_LINE);
          resample_kernel25<<<grid, block>>>(gpu_w, b_varargin_1, i, gpu_y);
        }
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        if (!(div_s32(u1, b_varargin_1) + 1 < Ly)) {
          i = cpu_y->size[0];
          if (Ly < 1.0) {
            cpu_y->size[0] = 0;
          } else {
            cpu_y->size[0] = static_cast<int32_T>(Ly);
          }
          nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
          emxEnsureCapacity_creal_T(cpu_y, i, &oe_emlrtRTEI);
          w_outdatedOnCpu = true;
        }
        if (std::isnan(dlast)) {
          i = b_cpu_y->size[0] * b_cpu_y->size[1];
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_cpu_y, i, &ib_emlrtRTEI);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
          y_needsGpuEnsureCapacity = false;
          nvtxMarkA("#resample_kernel26#" MW_AT_LINE);
          resample_kernel26<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b_gpu_y);
        } else if (dlast < 1.0) {
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = 0;
        } else {
          i = b_cpu_y->size[0] * b_cpu_y->size[1];
          b_cpu_y->size[0] = 1;
          b_cpu_y->size[1] = static_cast<int32_T>(dlast - 1.0) + 1;
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(b_cpu_y, i, &ib_emlrtRTEI);
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(
              computeNumIters(static_cast<int32_T>(dlast - 1.0)), &grid, &block,
              2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
          y_needsGpuEnsureCapacity = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#resample_kernel28#" MW_AT_LINE);
            resample_kernel28<<<grid, block>>>(
                static_cast<int32_T>(dlast - 1.0), b_gpu_y);
          }
        }
        b_varargin_1 = cpu_varargout_1->size[0];
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(b_varargin_1 - 1), &grid,
                                &block, 2147483647U);
        if (w_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal_T(cpu_y, &gpu_y, true);
        }
        if (y_needsGpuEnsureCapacity) {
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(b_cpu_y, &b_gpu_y, true);
        }
        y_needsGpuEnsureCapacity = false;
        if (c_varargout_1_needsGpuEnsureCap) {
          nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
        }
        c_varargout_1_needsGpuEnsureCap = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#resample_kernel27#" MW_AT_LINE);
          resample_kernel27<<<grid, block>>>(
              gpu_y, b_gpu_y, expl_temp.filterDelay, exponent, b_varargin_1 - 1,
              *gpu_varargout_1, cpu_varargout_1->size[0U]);
        }
        *varargout_1_outdatedOnCpu = true;
      }
      profileLoopEnd();
      nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
      emxFree_real_T(&b_cpu_y);
      nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
      emxFree_creal_T(&cpu_y);
      nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
      emxFree_creal_T(&cpu_w);
      nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
      emxFree_creal_T(&yCol);
    } else {
      *varargout_1_outdatedOnCpu = false;
      *varargout_1_outdatedOnGpu = false;
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = expl_temp.x->size[0];
      cpu_varargout_1->size[1] = expl_temp.x->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &he_emlrtRTEI);
      profileLoopStart("resample_loop_1", __LINE__,
                       (expl_temp.x->size[0] * expl_temp.x->size[1] - 1) + 1,
                       "");
      for (i = 0; i < expl_temp.x->size[0] * expl_temp.x->size[1]; i++) {
        cpu_varargout_1->data[i] = expl_temp.x->data[i];
        *varargout_1_outdatedOnGpu = true;
      }
      profileLoopEnd();
      nvtxMarkA("#uniformResampleAlongFirstDim#" MW_AT_LINE);
      b_signal::internal::resample::uniformResampleAlongFirstDim(
          cpu_varargout_1, varargout_1_outdatedOnCpu, gpu_varargout_1,
          varargout_1_outdatedOnGpu, expl_temp.p, expl_temp.q,
          expl_temp.filterWithPadding, expl_temp.filterDelay);
      c_varargout_1_needsGpuEnsureCap = true;
    }
    if ((expl_temp.x->size[0] == 1) || (expl_temp.x->size[1] == 1)) {
      uint32_T outSize_idx_1;
      outSize_idx_1 = static_cast<uint32_T>(cpu_varargout_1->size[0]);
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = static_cast<int32_T>(expl_temp.inputSize[0]);
      cpu_varargout_1->size[1] = static_cast<int32_T>(outSize_idx_1);
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &ie_emlrtRTEI);
    } else {
      nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
      emxInit_creal_T(&b_cpu_varargout_1, 2, &le_emlrtRTEI, true);
      i = b_cpu_varargout_1->size[0] * b_cpu_varargout_1->size[1];
      b_cpu_varargout_1->size[0] = cpu_varargout_1->size[1];
      b_cpu_varargout_1->size[1] = cpu_varargout_1->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(b_cpu_varargout_1, i, &le_emlrtRTEI);
      i = cpu_varargout_1->size[0] - 1;
      b_varargin_1 = cpu_varargout_1->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i, b_varargin_1), &grid, &block,
                              2147483647U);
      if (c_varargout_1_needsGpuEnsureCap) {
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1,
                                     !*varargout_1_outdatedOnGpu);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(b_cpu_varargout_1, &b_gpu_varargout_1, true);
      if (*varargout_1_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(gpu_varargout_1, cpu_varargout_1);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#resample_kernel29#" MW_AT_LINE);
        resample_kernel29<<<grid, block>>>(
            *gpu_varargout_1, b_varargin_1, i, b_gpu_varargout_1,
            b_cpu_varargout_1->size[0U], cpu_varargout_1->size[0U]);
      }
      i = cpu_varargout_1->size[0] * cpu_varargout_1->size[1];
      cpu_varargout_1->size[0] = b_cpu_varargout_1->size[0];
      cpu_varargout_1->size[1] = b_cpu_varargout_1->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_varargout_1, i, &ie_emlrtRTEI);
      i = b_cpu_varargout_1->size[0] * b_cpu_varargout_1->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_varargout_1, gpu_varargout_1, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#resample_kernel30#" MW_AT_LINE);
        resample_kernel30<<<grid, block>>>(b_gpu_varargout_1, i,
                                           *gpu_varargout_1);
      }
      *varargout_1_outdatedOnGpu = false;
      *varargout_1_outdatedOnCpu = true;
      nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
      emxFree_creal_T(&b_cpu_varargout_1);
    }
  }
  nvtxMarkA("#emxFreeStruct_ResampleParser#" MW_AT_LINE);
  emxFreeStruct_ResampleParser(&expl_temp);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_opts_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_varargout_1);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_w);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_y);
  *varargin_1_outdatedOnGpu = false;
  nvtxRangePop();
}

} // namespace coder

// End of code generation (resample.cu)

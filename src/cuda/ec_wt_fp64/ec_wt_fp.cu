#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp.cu
//
// Code generation for function 'ec_wt_fp'
//

// Include files
#include "ec_wt_fp.h"
#include "cwtfilterbank.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "rt_nonfinite.h"
#include "scaleSpectrum.h"
#include "wt.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtMCInfo c_emlrtMCI{
    53,        // lineNo
    19,        // colNo
    "flt2str", // fName
    "/usr/local/MATLAB/R2024a/toolbox/shared/coder/coder/lib/+coder/+internal/"
    "flt2str.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    30,                                                     // lineNo
    1,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    39,                                                     // lineNo
    24,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    41,                                                     // lineNo
    24,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    47,                                                     // lineNo
    30,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    49,                                                     // lineNo
    32,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    47,                                                     // lineNo
    19,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    63,                                                     // lineNo
    14,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    61,                                                     // lineNo
    14,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    75,                                                           // lineNo
    13,                                                           // colNo
    "downsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/downsample.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    287,             // lineNo
    33,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    91,                                                           // lineNo
    14,                                                           // colNo
    "downsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/downsample.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    181,                     // lineNo
    24,                      // colNo
    "combineVectorElements", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
    "combineVectorElements.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    49,                                                     // lineNo
    21,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    287,             // lineNo
    25,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    28,                                                     // lineNo
    1,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    61,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    47,                                                     // lineNo
    9,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    267,             // lineNo
    36,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    49,                                                     // lineNo
    9,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    282,             // lineNo
    9,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    287,             // lineNo
    29,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo ab_emlrtRTEI{
    78,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23]);

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location);

static uint64_T computeNumIters(int32_T ub, int32_T b_ub);

static
#ifdef __HIPCC__
    __device__
#endif
        int32_T
        div_s32_device(int32_T numerator, int32_T denominator);

static __global__ void ec_wt_fp_kernel1(const int32_T ch,
                                        const emxArray_real_T x,
                                        const int32_T b_x, emxArray_real_T xc,
                                        int32_T x_dim0);

static __global__ void ec_wt_fp_kernel10(const emxArray_real_T b_dv,
                                         const real_T ds, const int32_T b,
                                         emxArray_real_T c_dv);

static __global__ void ec_wt_fp_kernel11(const emxArray_real_T b_dv,
                                         const int32_T c_dv,
                                         emxArray_real_T d_dv);

static __global__ void ec_wt_fp_kernel12(const emxArray_real_T b_dv,
                                         const int32_T ch,
                                         const int32_T lshift_dim0,
                                         const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel13(const int32_T vlen,
                                         const emxArray_real_T x,
                                         const int32_T b, emxArray_real_T b_b,
                                         int32_T x_dim0);

static __global__ void ec_wt_fp_kernel14(const emxArray_real_T r,
                                         const int32_T b_r,
                                         emxArray_real_T b_dv);

static __global__ void ec_wt_fp_kernel15(const int32_T ch,
                                         const emxArray_real_T x,
                                         const int32_T b_x, emxArray_real_T xc,
                                         int32_T x_dim0);

static __global__ void ec_wt_fp_kernel16(const emxArray_creal_T r2,
                                         const int32_T b_r2, const int32_T c_r2,
                                         emxArray_creal_T x, int32_T x_dim0,
                                         int32_T r2_dim0);

static __global__ void ec_wt_fp_kernel17(const emxArray_creal_T x,
                                         const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel18(const emxArray_real_T y,
                                         const int32_T b, emxArray_real_T dv4);

static __global__ void ec_wt_fp_kernel19(const int32_T shiftLen,
                                         const int32_T b, int32_T lshift[2]);

static __global__ void ec_wt_fp_kernel2(const emxArray_creal_T cfs,
                                        const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel20(const int32_T vlen,
                                         const int32_T shiftLen,
                                         const int32_T dim, int32_T sizeX[2]);

static __global__ void ec_wt_fp_kernel21(const emxArray_real_T dv4,
                                         const int32_T lshift_dim0,
                                         const int32_T vlen, const int32_T b,
                                         const int32_T c, emxArray_real_T b_dv4,
                                         int32_T dv4_dim0);

static __global__ void ec_wt_fp_kernel22(const emxArray_real_T dv4,
                                         const int32_T lshift,
                                         emxArray_real_T b_dv4);

static __global__ void ec_wt_fp_kernel23(const emxArray_real_T dv4,
                                         const int32_T y_dim0, const int32_T ch,
                                         const int32_T y, const int32_T b_y,
                                         emxArray_real_T c_y, int32_T b_y_dim0,
                                         int32_T y_dim1);

static __global__ void ec_wt_fp_kernel24(const emxArray_creal_T r1,
                                         const int32_T b_r1, const int32_T c_r1,
                                         emxArray_creal_T x, int32_T x_dim0,
                                         int32_T r1_dim0);

static __global__ void ec_wt_fp_kernel25(const emxArray_creal_T x,
                                         const int32_T b, emxArray_real_T dv4);

static __global__ void ec_wt_fp_kernel3(const emxArray_real_T y,
                                        const int32_T b, emxArray_real_T x);

static __global__ void ec_wt_fp_kernel4(const real_T nFrames,
                                        const emxArray_creal_T cfs,
                                        const int32_T nx, const int32_T cfsnorm,
                                        const int32_T vlen,
                                        emxArray_creal_T b_cfsnorm,
                                        int32_T cfs_dim0, int32_T cfsnorm_dim0);

static __global__ void ec_wt_fp_kernel5(const emxArray_creal_T cfsnorm,
                                        const int32_T b,
                                        const int32_T b_cfsnorm,
                                        emxArray_creal_T x, int32_T x_dim0,
                                        int32_T cfsnorm_dim0);

static __global__ void ec_wt_fp_kernel6(const emxArray_creal_T x,
                                        const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel7(const emxArray_real_T y,
                                        const int32_T b, emxArray_real_T x);

static __global__ void ec_wt_fp_kernel8(const int32_T sz, emxArray_real_T b);

static __global__ void ec_wt_fp_kernel9(const emxArray_real_T b,
                                        const real_T nFrames, const int32_T b_b,
                                        emxArray_real_T b_dv);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23]);

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23]);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[23])
{
  static const int32_T dims[2]{1, 23};
  nvtxRangePushA("#fcn#b_emlrt_marshallIn#" MW_AT_LOCATION);
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                          (const void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 23);
  emlrtDestroyArray(&src);
  nvtxRangePop();
}

static const mxArray *b_sprintf(const mxArray *m1, const mxArray *m2,
                                emlrtMCInfo *location)
{
  const mxArray *pArrays[2];
  const mxArray *m;
  const mxArray *m3;
  nvtxRangePushA("#fcn#b_sprintf#" MW_AT_LOCATION);
  pArrays[0] = m1;
  pArrays[1] = m2;
  m3 = emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 2, &pArrays[0],
                             "sprintf", true, location);
  nvtxRangePop();
  return m3;
}

static uint64_T computeNumIters(int32_T ub, int32_T b_ub)
{
  uint64_T n;
  uint64_T numIters;
  nvtxRangePushA("#fcn#computeNumIters#" MW_AT_LOCATION);
  n = 0UL;
  if (ub >= 0) {
    n = static_cast<uint64_T>(ub + 1);
  }
  numIters = n;
  n = 0UL;
  if (b_ub >= 0) {
    n = static_cast<uint64_T>(b_ub + 1);
  }
  numIters *= n;
  nvtxRangePop();
  return numIters;
}

static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if (static_cast<int32_T>(numerator < 0) !=
        static_cast<int32_T>(denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel1(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T xc, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    xc.data[xpageoffset] = x.data[xpageoffset + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel10(
    const emxArray_real_T b_dv, const real_T ds, const int32_T b,
    emxArray_real_T c_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    c_dv.data[xpageoffset] = b_dv.data[static_cast<int32_T>(ds) * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel11(
    const emxArray_real_T b_dv, const int32_T c_dv, emxArray_real_T d_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    d_dv.data[xpageoffset] = b_dv.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel12(
    const emxArray_real_T b_dv, const int32_T ch, const int32_T lshift_dim0,
    const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    y.data[xpageoffset + lshift_dim0 * ch] = b_dv.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel13(
    const int32_T vlen, const emxArray_real_T x, const int32_T b,
    emxArray_real_T b_b, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx);
    xpageoffset = bcoef * x_dim0;
    b_b.data[bcoef] = x.data[xpageoffset];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      b_b.data[bcoef] += x.data[(xpageoffset + k) + 1];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel14(
    const emxArray_real_T r, const int32_T b_r, emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_dv.data[xpageoffset] = r.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel15(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T xc, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    xc.data[xpageoffset] = x.data[xpageoffset + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel16(
    const emxArray_creal_T r2, const int32_T b_r2, const int32_T c_r2,
    emxArray_creal_T x, int32_T x_dim0, int32_T r2_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r2) + 1UL) *
                (static_cast<uint64_T>(b_r2) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r2) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b_r2) + 1UL));
    x.data[bcoef + x_dim0 * xpageoffset].re =
        r2.data[xpageoffset + r2_dim0 * bcoef].re;
    x.data[bcoef + x_dim0 * xpageoffset].im =
        -r2.data[xpageoffset + r2_dim0 * bcoef].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel17(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel18(
    const emxArray_real_T y, const int32_T b, emxArray_real_T dv4)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    dv4.data[k] = y.data[k] * y.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel19(
    const int32_T shiftLen, const int32_T b, int32_T lshift[2])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    bcoef = lshift[0];
    lshift[0] = lshift[1 - shiftLen];
    lshift[1 - shiftLen] = bcoef;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel2(
    const emxArray_creal_T cfs, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(cfs.data[k].re, cfs.data[k].im);
  }
}

static __global__ __launch_bounds__(32, 1) void ec_wt_fp_kernel20(
    const int32_T vlen, const int32_T shiftLen, const int32_T dim,
    int32_T sizeX[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    sizeX[dim - 1] = div_s32_device(shiftLen, vlen) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel21(
    const emxArray_real_T dv4, const int32_T lshift_dim0, const int32_T vlen,
    const int32_T b, const int32_T c, emxArray_real_T b_dv4, int32_T dv4_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b) + 1UL));
    b_dv4.data[bcoef + dv4_dim0 * xpageoffset] =
        dv4.data[vlen * bcoef + lshift_dim0 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel22(
    const emxArray_real_T dv4, const int32_T lshift, emxArray_real_T b_dv4)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(lshift);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_dv4.data[xpageoffset] = dv4.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel23(
    const emxArray_real_T dv4, const int32_T y_dim0, const int32_T ch,
    const int32_T y, const int32_T b_y, emxArray_real_T c_y, int32_T b_y_dim0,
    int32_T y_dim1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b_y) + 1UL) * (static_cast<uint64_T>(y) + 1UL) -
      1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(y) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(y) + 1UL));
    c_y.data[(bcoef + b_y_dim0 * ch) + b_y_dim0 * y_dim1 * xpageoffset] =
        dv4.data[bcoef + y_dim0 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel24(
    const emxArray_creal_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_creal_T x, int32_T x_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b_r1) + 1UL));
    x.data[bcoef + x_dim0 * xpageoffset].re =
        r1.data[xpageoffset + r1_dim0 * bcoef].re;
    x.data[bcoef + x_dim0 * xpageoffset].im =
        -r1.data[xpageoffset + r1_dim0 * bcoef].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel25(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T dv4)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    dv4.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel3(
    const emxArray_real_T y, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x.data[k] = y.data[k] * y.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel4(
    const real_T nFrames, const emxArray_creal_T cfs, const int32_T nx,
    const int32_T cfsnorm, const int32_T vlen, emxArray_creal_T b_cfsnorm,
    int32_T cfs_dim0, int32_T cfsnorm_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(vlen) + 1UL) *
                (static_cast<uint64_T>(cfsnorm) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    int32_T bcoef;
    int32_T k;
    int32_T xpageoffset;
    b_k = static_cast<int32_T>(idx % (static_cast<uint64_T>(cfsnorm) + 1UL));
    k = static_cast<int32_T>((idx - static_cast<uint64_T>(b_k)) /
                             (static_cast<uint64_T>(cfsnorm) + 1UL));
    xpageoffset = nx * k + 1;
    bcoef = static_cast<int32_T>(cfs_dim0 != 1);
    b_cfsnorm.data[b_k + cfsnorm_dim0 * k].re =
        nFrames * cfs.data[bcoef * b_k + cfs_dim0 * (xpageoffset - 1)].re;
    b_cfsnorm.data[b_k + cfsnorm_dim0 * k].im =
        nFrames * cfs.data[bcoef * b_k + cfs_dim0 * (xpageoffset - 1)].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel5(
    const emxArray_creal_T cfsnorm, const int32_T b, const int32_T b_cfsnorm,
    emxArray_creal_T x, int32_T x_dim0, int32_T cfsnorm_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_cfsnorm) + 1UL) *
                (static_cast<uint64_T>(b) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b) + 1UL));
    x.data[bcoef + x_dim0 * xpageoffset] =
        cfsnorm.data[bcoef + cfsnorm_dim0 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel6(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel7(
    const emxArray_real_T y, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x.data[k] = y.data[k] * y.data[k];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void ec_wt_fp_kernel8(const int32_T sz,
                                                             emxArray_real_T b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel9(
    const emxArray_real_T b, const real_T nFrames, const int32_T b_b,
    emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_dv.data[xpageoffset] = nFrames * b.data[xpageoffset];
  }
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[23])
{
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  nvtxMarkA("#b_emlrt_marshallIn#" MW_AT_LINE);
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
  nvtxRangePop();
}

static void emlrt_marshallIn(const mxArray *a__output_of_sprintf_,
                             const char_T *identifier, char_T y[23])
{
  emlrtMsgIdentifier thisId;
  nvtxRangePushA("#fcn#emlrt_marshallIn#" MW_AT_LOCATION);
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
  emlrt_marshallIn(emlrtAlias(a__output_of_sprintf_), &thisId, y);
  emlrtDestroyArray(&a__output_of_sprintf_);
  nvtxRangePop();
}

//
// function [y,freqs] = ec_wt_fp(x,fs,fLims,fVoices,ds,doAvg,doPwr)
void ec_wt_fp(const emxArray_real_T *cpu_x, real_T fs, const real_T fLims[2],
              real_T fVoices, real_T ds, boolean_T doAvg, boolean_T doPwr,
              emxArray_real_T *cpu_y, emxArray_real_T *freqs)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  coder::cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal_T c_gpu_x;
  emxArray_creal_T e_gpu_x;
  emxArray_creal_T f_gpu_x;
  emxArray_creal_T gpu_cfs;
  emxArray_creal_T gpu_cfsnorm;
  emxArray_creal_T gpu_r1;
  emxArray_creal_T gpu_r2;
  emxArray_creal_T *b_cpu_x;
  emxArray_creal_T *c_cpu_x;
  emxArray_creal_T *cpu_cfs;
  emxArray_creal_T *cpu_cfsnorm;
  emxArray_creal_T *cpu_r1;
  emxArray_creal_T *cpu_r2;
  emxArray_creal_T *e_cpu_x;
  emxArray_real_T b_gpu_dv;
  emxArray_real_T b_gpu_dv4;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_xc;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_x;
  emxArray_real_T d_gpu_y;
  emxArray_real_T gpu_b;
  emxArray_real_T gpu_dv;
  emxArray_real_T gpu_dv4;
  emxArray_real_T gpu_r;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xc;
  emxArray_real_T gpu_y;
  emxArray_real_T *b_cpu_dv;
  emxArray_real_T *b_cpu_dv4;
  emxArray_real_T *b_cpu_xc;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_b;
  emxArray_real_T *cpu_dv;
  emxArray_real_T *cpu_dv4;
  emxArray_real_T *cpu_r;
  emxArray_real_T *cpu_xc;
  emxArray_real_T *d_cpu_x;
  emxArray_real_T *d_cpu_y;
  emxArray_real_T *f_cpu_x;
  const mxArray *b_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T nFrames;
  int32_T(*gpu_lshift)[2];
  int32_T(*gpu_sizeX)[2];
  int32_T dim;
  int32_T i;
  char_T unusedExpr[23];
  boolean_T r_outdatedOnCpu;
  boolean_T r_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_wt_fp#" MW_AT_LOCATION);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_lshift, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaMalloc#" MW_AT_LINE);
  checkCudaError(mwCudaMalloc(&gpu_sizeX, 8UL), __FILE__, __LINE__);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_dv4);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_dv);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r2);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r1);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_b);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfsnorm);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv4);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&e_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&f_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfs);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_xc);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xc);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x);
  r_outdatedOnGpu = false;
  x_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInitStruct_cwtfilterbank#" MW_AT_LINE);
  emxInitStruct_cwtfilterbank(&fb, &r_emlrtRTEI, true);
  //  electroCUDA - run Continuous WAvelet Transform
  //    Intended to be compiled into a CUDA mex binary
  //    Kevin Tan, 2024 (github.com/kevmtan/electroCUDA)
  //
  //  OUTPUTS:
  //    y = transformed data
  //    freqs = CWT frequencies
  // 'ec_wt_fp:10' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_wt_fp:11' fs (1,1) double
  //  Sampling rate
  // 'ec_wt_fp:12' fLims (1,2) double
  //  Frequency limits
  // 'ec_wt_fp:13' fVoices (1,1) double = 10
  //  Voices per octave
  // 'ec_wt_fp:14' ds (1,1) double = 0
  //  Downsampling factor
  // 'ec_wt_fp:15' doAvg (1,1) logical = false
  //  Transform [0=continuous|1=averaged]
  // 'ec_wt_fp:16' doPwr (1,1) logical = false
  //  Output [0=magnitude|1=output]
  // 'ec_wt_fp:18' if ds<=1
  if (ds <= 1.0) {
    // 'ec_wt_fp:18' ;
    // 'ec_wt_fp:18' ds=0;
    ds = 0.0;
  }
  //  Prep
  // 'ec_wt_fp:21' coder.gpu.kernelfun;
  //  Trigger CUDA kernel creation
  //  Sizes
  // 'ec_wt_fp:24' nFrames = height(x);
  nFrames = cpu_x->size[0];
  // 'ec_wt_fp:25' nChs = width(x);
  //  Make CWT filter
  // 'ec_wt_fp:28' fb = cwtfilterbank(Wavelet="Morse",SignalLength=nFrames,...
  // 'ec_wt_fp:29'
  // SamplingFrequency=fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
  nvtxMarkA("#cwtfilterbank_cwtfilterbank#" MW_AT_LINE);
  coder::cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(cpu_x->size[0]),
                                     fs, fLims, fVoices);
  // 'ec_wt_fp:30' freqs = centerFrequencies(fb);
  dim = freqs->size[0];
  freqs->size[0] = fb.WaveletCenterFrequencies->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(freqs, dim, &emlrtRTEI);
  profileLoopStart("ec_wt_fp_loop_0", __LINE__,
                   (fb.WaveletCenterFrequencies->size[0] - 1) + 1, "");
  for (dim = 0; dim < fb.WaveletCenterFrequencies->size[0]; dim++) {
    freqs->data[dim] = fb.WaveletCenterFrequencies->data[dim];
  }
  profileLoopEnd();
  //  Find output frames
  // 'ec_wt_fp:33' if ds
  if (ds != 0.0) {
    // 'ec_wt_fp:34' nFrames = floor(nFrames/ds);
    nFrames = std::floor(static_cast<real_T>(cpu_x->size[0]) / ds);
  }
  //  Preallocate output
  // 'ec_wt_fp:38' if doAvg
  if (doAvg) {
    // 'ec_wt_fp:39' y = coder.nullcopy(nan(nFrames,nChs,like=x));
    r_outdatedOnCpu = false;
    dim = cpu_y->size[0] * cpu_y->size[1] * cpu_y->size[2];
    cpu_y->size[0] = static_cast<int32_T>(nFrames);
    cpu_y->size[1] = cpu_x->size[1];
    cpu_y->size[2] = 1;
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, dim, &b_emlrtRTEI);
    y_needsGpuEnsureCapacity = true;
  } else {
    // 'ec_wt_fp:40' else
    // 'ec_wt_fp:41' y = coder.nullcopy(nan(nFrames,nChs,numel(freqs),like=x));
    r_outdatedOnCpu = false;
    dim = cpu_y->size[0] * cpu_y->size[1] * cpu_y->size[2];
    cpu_y->size[0] = static_cast<int32_T>(nFrames);
    cpu_y->size[1] = cpu_x->size[1];
    cpu_y->size[2] = freqs->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_y, dim, &c_emlrtRTEI);
    y_needsGpuEnsureCapacity = true;
  }
  //  Processing loop across channels
  // 'ec_wt_fp:45' for ch = 1:nChs
  i = cpu_x->size[1];
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xc, 1, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_xc, 1, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfs, 2, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_y, 2, &t_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv, 1, &u_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_x, 2, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&c_cpu_x, 2, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_y, 2, &v_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv4, 2, &w_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_x, 2, &v_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfsnorm, 2, &x_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&e_cpu_x, 2, &m_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_y, 2, &y_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&f_cpu_x, 2, &y_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_b, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_r, 2, &ab_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r1, 2, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r2, 2, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_dv, 1, &k_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_dv4, 2, &n_emlrtRTEI, true);
  profileLoopStart("ec_wt_fp_loop_1", __LINE__, (i - 1) + 1, "");
  for (int32_T ch{0}; ch < i; ch++) {
    // 'ec_wt_fp:46' if doAvg
    if (doAvg) {
      int32_T lshift[2];
      int32_T nx;
      int32_T xpageoffset;
      boolean_T validLaunchParams;
      // 'ec_wt_fp:47' y(:,ch) = awt_lfn(fb,x(:,ch),ds,doPwr);
      dim = cpu_xc->size[0];
      cpu_xc->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_xc, dim, &d_emlrtRTEI);
      xpageoffset = cpu_x->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      if (x_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
      }
      x_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_xc, &gpu_xc, true);
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
      }
      x_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel1#" MW_AT_LINE);
        ec_wt_fp_kernel1<<<grid, block>>>(ch, gpu_x, xpageoffset, gpu_xc,
                                          cpu_x->size[0U]);
      }
      //  Scale-averaged wavelet transform
      //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%% Run transform
      // 'ec_wt_fp:75' if doPwr
      if (doPwr) {
        int32_T vlen;
        uint32_T sz[2];
        // 'ec_wt_fp:76' yc = scaleSpectrum(fb,xc)';
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
        nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
        coder::cwtfilterbank_wt(&fb, cpu_xc, cpu_cfs);
        nx = cpu_cfs->size[0] * cpu_cfs->size[1];
        profileLoopStart("ec_wt_fp_loop_2", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(cpu_cfs->size[xpageoffset]);
        }
        profileLoopEnd();
        dim = c_cpu_y->size[0] * c_cpu_y->size[1];
        c_cpu_y->size[0] = static_cast<int32_T>(sz[0]);
        c_cpu_y->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(c_cpu_y, dim, &i_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_cfs, &gpu_cfs, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(c_cpu_y, &gpu_y, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_cfs, cpu_cfs);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel2#" MW_AT_LINE);
          ec_wt_fp_kernel2<<<grid, block>>>(gpu_cfs, nx - 1, gpu_y);
        }
        profileLoopStart("ec_wt_fp_loop_5", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(c_cpu_y->size[xpageoffset]);
        }
        profileLoopEnd();
        dim = d_cpu_x->size[0] * d_cpu_x->size[1];
        d_cpu_x->size[0] = static_cast<int32_T>(sz[0]);
        d_cpu_x->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(d_cpu_x, dim, &j_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(
                static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
            &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(d_cpu_x, &b_gpu_x, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel3#" MW_AT_LINE);
          ec_wt_fp_kernel3<<<grid, block>>>(
              gpu_y,
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
              b_gpu_x);
        }
        vlen = d_cpu_x->size[0] * d_cpu_x->size[1];
        if (d_cpu_x->size[0] * d_cpu_x->size[1] == 0) {
          nFrames = 0.0;
        } else {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_real_T(d_cpu_x, &b_gpu_x);
          nFrames = d_cpu_x->data[0];
          profileLoopStart("ec_wt_fp_loop_7", __LINE__, (vlen - 2) + 1, "");
          for (nx = 0; nx <= vlen - 2; nx++) {
            nFrames += d_cpu_x->data[nx + 1];
          }
          profileLoopEnd();
        }
        nFrames = std::sqrt(
            fb.sigvar /
            (1.0 / static_cast<real_T>(fb.Scales->size[1]) * nFrames));
        dim = cpu_cfsnorm->size[0] * cpu_cfsnorm->size[1];
        cpu_cfsnorm->size[0] = cpu_cfs->size[0];
        cpu_cfsnorm->size[1] = cpu_cfs->size[1];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(cpu_cfsnorm, dim, &l_emlrtRTEI);
        r_outdatedOnCpu = true;
        if ((cpu_cfs->size[0] != 0) && (cpu_cfs->size[1] != 0)) {
          nx = (cpu_cfs->size[1] != 1);
          vlen = cpu_cfs->size[1] - 1;
          xpageoffset = cpu_cfsnorm->size[0] - 1;
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(vlen, xpageoffset), &grid,
                                  &block, 2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal_T(cpu_cfsnorm, &gpu_cfsnorm, true);
          r_outdatedOnCpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wt_fp_kernel4#" MW_AT_LINE);
            ec_wt_fp_kernel4<<<grid, block>>>(
                nFrames, gpu_cfs, nx, xpageoffset, vlen, gpu_cfsnorm,
                cpu_cfs->size[0U], cpu_cfsnorm->size[0U]);
          }
        }
        vlen = fb.Scales->size[1];
        nFrames = 1.0 / static_cast<real_T>(fb.Scales->size[1]);
        dim = e_cpu_x->size[0] * e_cpu_x->size[1];
        e_cpu_x->size[0] = vlen;
        e_cpu_x->size[1] = cpu_cfsnorm->size[1];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(e_cpu_x, dim, &m_emlrtRTEI);
        xpageoffset = cpu_cfsnorm->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset, vlen - 1), &grid,
                                &block, 2147483647U);
        if (r_outdatedOnCpu) {
          nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_creal_T(cpu_cfsnorm, &gpu_cfsnorm, true);
        }
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(e_cpu_x, &c_gpu_x, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel5#" MW_AT_LINE);
          ec_wt_fp_kernel5<<<grid, block>>>(gpu_cfsnorm, vlen - 1, xpageoffset,
                                            c_gpu_x, e_cpu_x->size[0U],
                                            cpu_cfsnorm->size[0U]);
        }
        nx = vlen * cpu_cfsnorm->size[1];
        dim = d_cpu_y->size[0] * d_cpu_y->size[1];
        d_cpu_y->size[0] = vlen;
        d_cpu_y->size[1] = cpu_cfsnorm->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(d_cpu_y, dim, &i_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(d_cpu_y, &b_gpu_y, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel6#" MW_AT_LINE);
          ec_wt_fp_kernel6<<<grid, block>>>(c_gpu_x, nx - 1, b_gpu_y);
        }
        profileLoopStart("ec_wt_fp_loop_9", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(d_cpu_y->size[xpageoffset]);
        }
        profileLoopEnd();
        dim = f_cpu_x->size[0] * f_cpu_x->size[1];
        f_cpu_x->size[0] = static_cast<int32_T>(sz[0]);
        f_cpu_x->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(f_cpu_x, dim, &j_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(
                static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
            &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(f_cpu_x, &d_gpu_x, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel7#" MW_AT_LINE);
          ec_wt_fp_kernel7<<<grid, block>>>(
              b_gpu_y,
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
              d_gpu_x);
        }
        vlen = f_cpu_x->size[0];
        if ((f_cpu_x->size[0] == 0) || (f_cpu_x->size[1] == 0)) {
          profileLoopStart("ec_wt_fp_loop_10", __LINE__, 1 + 1, "");
          for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
            sz[xpageoffset] = static_cast<uint32_T>(f_cpu_x->size[xpageoffset]);
          }
          profileLoopEnd();
          dim = cpu_b->size[0] * cpu_b->size[1];
          cpu_b->size[0] = 1;
          cpu_b->size[1] = static_cast<int32_T>(sz[1]);
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_b, dim, &q_emlrtRTEI);
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(
              computeNumIters(static_cast<int32_T>(sz[1]) - 1), &grid, &block,
              2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wt_fp_kernel8#" MW_AT_LINE);
            ec_wt_fp_kernel8<<<grid, block>>>(static_cast<int32_T>(sz[1]) - 1,
                                              gpu_b);
          }
        } else {
          nx = f_cpu_x->size[1];
          dim = cpu_b->size[0] * cpu_b->size[1];
          cpu_b->size[0] = 1;
          cpu_b->size[1] = f_cpu_x->size[1];
          nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
          emxEnsureCapacity_real_T(cpu_b, dim, &o_emlrtRTEI);
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                  2147483647U);
          nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
          gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wt_fp_kernel13#" MW_AT_LINE);
            ec_wt_fp_kernel13<<<grid, block>>>(vlen, d_gpu_x, nx - 1, gpu_b,
                                               f_cpu_x->size[0U]);
          }
        }
        dim = cpu_dv->size[0];
        cpu_dv->size[0] = cpu_b->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_dv, dim, &f_emlrtRTEI);
        xpageoffset = cpu_b->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel9#" MW_AT_LINE);
          ec_wt_fp_kernel9<<<grid, block>>>(gpu_b, nFrames, xpageoffset,
                                            gpu_dv);
        }
        //  Power
      } else {
        // 'ec_wt_fp:77' else
        // 'ec_wt_fp:78' yc = scaleSpectrum(fb,xc,SpectrumType="density")';
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
        nvtxMarkA("#cwtfilterbank_scaleSpectrum#" MW_AT_LINE);
        coder::cwtfilterbank_scaleSpectrum(&fb, cpu_xc, cpu_r, &r_outdatedOnCpu,
                                           &gpu_r, &r_outdatedOnGpu);
        dim = cpu_dv->size[0];
        cpu_dv->size[0] = cpu_r->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_dv, dim, &f_emlrtRTEI);
        xpageoffset = cpu_r->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_r, &gpu_r, !r_outdatedOnGpu);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
        if (r_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
          gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, cpu_r);
        }
        r_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel14#" MW_AT_LINE);
          ec_wt_fp_kernel14<<<grid, block>>>(gpu_r, xpageoffset, gpu_dv);
        }
        //  Magnitude
      }
      //  Downsample
      // 'ec_wt_fp:82' if ds
      if (ds != 0.0) {
        // 'ec_wt_fp:83' yc = downsample(yc,ds);
        if (!(ds - 1.0 >= 0.0)) {
          y = nullptr;
          m = emlrtCreateCharArray(2, &iv[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
          emlrtAssign(&y, m);
          b_y = nullptr;
          m1 = emlrtCreateDoubleScalar(ds - 1.0);
          emlrtAssign(&b_y, m1);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(y, b_y, &c_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        nx = cpu_dv->size[0] - 1;
        dim = b_cpu_dv->size[0];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        b_cpu_dv->size[0] =
            div_s32(cpu_dv->size[0] - 1, static_cast<int32_T>(ds)) + 1;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_dv, dim, &k_emlrtRTEI);
        xpageoffset = nx / static_cast<int32_T>(ds);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_dv, &b_gpu_dv, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel10#" MW_AT_LINE);
          ec_wt_fp_kernel10<<<grid, block>>>(gpu_dv, ds, xpageoffset, b_gpu_dv);
        }
        dim = cpu_dv->size[0];
        cpu_dv->size[0] = b_cpu_dv->size[0];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_dv, dim, &f_emlrtRTEI);
        xpageoffset = b_cpu_dv->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel11#" MW_AT_LINE);
          ec_wt_fp_kernel11<<<grid, block>>>(b_gpu_dv, xpageoffset, gpu_dv);
        }
      }
      nx = cpu_y->size[0];
      lshift[0] = cpu_y->size[0];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      if (y_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_y, &c_gpu_y, true);
      }
      y_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel12#" MW_AT_LINE);
        ec_wt_fp_kernel12<<<grid, block>>>(gpu_dv, ch, lshift[0], nx - 1,
                                           c_gpu_y);
      }
      r_outdatedOnCpu = true;
    } else {
      int32_T c_y[3];
      int32_T nx;
      int32_T vlen;
      int32_T xpageoffset;
      boolean_T validLaunchParams;
      // 'ec_wt_fp:48' else
      // 'ec_wt_fp:49' y(:,ch,:) = cwt_lfn(fb,x(:,ch),ds,doPwr);
      dim = b_cpu_xc->size[0];
      b_cpu_xc->size[0] = cpu_x->size[0];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_xc, dim, &e_emlrtRTEI);
      xpageoffset = cpu_x->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      if (x_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
      }
      x_needsGpuEnsureCapacity = false;
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_xc, &b_gpu_xc, true);
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
      }
      x_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel15#" MW_AT_LINE);
        ec_wt_fp_kernel15<<<grid, block>>>(ch, gpu_x, xpageoffset, b_gpu_xc,
                                           cpu_x->size[0U]);
      }
      //  Continuous wavelet transform
      //  %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%% Run transform
      // 'ec_wt_fp:60' if doPwr
      if (doPwr) {
        uint32_T sz[2];
        // 'ec_wt_fp:61' yc = abs(wt(fb,xc)').^2;
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(b_cpu_xc, &b_gpu_xc);
        nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
        coder::cwtfilterbank_wt(&fb, b_cpu_xc, cpu_r2);
        dim = c_cpu_x->size[0] * c_cpu_x->size[1];
        c_cpu_x->size[0] = cpu_r2->size[1];
        c_cpu_x->size[1] = cpu_r2->size[0];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(c_cpu_x, dim, &h_emlrtRTEI);
        xpageoffset = cpu_r2->size[0] - 1;
        vlen = cpu_r2->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset, vlen), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_r2, &gpu_r2, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(c_cpu_x, &e_gpu_x, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r2, cpu_r2);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel16#" MW_AT_LINE);
          ec_wt_fp_kernel16<<<grid, block>>>(gpu_r2, vlen, xpageoffset, e_gpu_x,
                                             c_cpu_x->size[0U],
                                             cpu_r2->size[0U]);
        }
        nx = c_cpu_x->size[0] * c_cpu_x->size[1];
        profileLoopStart("ec_wt_fp_loop_3", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(c_cpu_x->size[xpageoffset]);
        }
        profileLoopEnd();
        dim = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = static_cast<int32_T>(sz[0]);
        b_cpu_y->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_y, dim, &i_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_y, &d_gpu_y, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel17#" MW_AT_LINE);
          ec_wt_fp_kernel17<<<grid, block>>>(e_gpu_x, nx - 1, d_gpu_y);
        }
        profileLoopStart("ec_wt_fp_loop_6", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(b_cpu_y->size[xpageoffset]);
        }
        profileLoopEnd();
        dim = cpu_dv4->size[0] * cpu_dv4->size[1];
        cpu_dv4->size[0] = static_cast<int32_T>(sz[0]);
        cpu_dv4->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_dv4, dim, &j_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(
                static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
            &grid, &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_dv4, &gpu_dv4, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel18#" MW_AT_LINE);
          ec_wt_fp_kernel18<<<grid, block>>>(
              d_gpu_y,
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
              gpu_dv4);
        }
        //  Power
      } else {
        uint32_T sz[2];
        // 'ec_wt_fp:62' else
        // 'ec_wt_fp:63' yc = abs(wt(fb,xc)');
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(b_cpu_xc, &b_gpu_xc);
        nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
        coder::cwtfilterbank_wt(&fb, b_cpu_xc, cpu_r1);
        dim = b_cpu_x->size[0] * b_cpu_x->size[1];
        b_cpu_x->size[0] = cpu_r1->size[1];
        b_cpu_x->size[1] = cpu_r1->size[0];
        nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
        emxEnsureCapacity_creal_T(b_cpu_x, dim, &g_emlrtRTEI);
        xpageoffset = cpu_r1->size[0] - 1;
        vlen = cpu_r1->size[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset, vlen), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_r1, &gpu_r1, false);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(b_cpu_x, &f_gpu_x, true);
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r1, cpu_r1);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel24#" MW_AT_LINE);
          ec_wt_fp_kernel24<<<grid, block>>>(gpu_r1, vlen, xpageoffset, f_gpu_x,
                                             b_cpu_x->size[0U],
                                             cpu_r1->size[0U]);
        }
        nx = b_cpu_x->size[0] * b_cpu_x->size[1];
        profileLoopStart("ec_wt_fp_loop_4", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(b_cpu_x->size[xpageoffset]);
        }
        profileLoopEnd();
        dim = cpu_dv4->size[0] * cpu_dv4->size[1];
        cpu_dv4->size[0] = static_cast<int32_T>(sz[0]);
        cpu_dv4->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_dv4, dim, &i_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_dv4, &gpu_dv4, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel25#" MW_AT_LINE);
          ec_wt_fp_kernel25<<<grid, block>>>(f_gpu_x, nx - 1, gpu_dv4);
        }
        //  Magnitude
      }
      //  Downsample
      // 'ec_wt_fp:67' if ds
      if (ds != 0.0) {
        int32_T cpu_lshift[2];
        int32_T cpu_sizeX[2];
        // 'ec_wt_fp:68' yc = downsample(yc,ds);
        if (!(ds - 1.0 >= 0.0)) {
          d_y = nullptr;
          m2 = emlrtCreateCharArray(2, &iv1[0]);
          emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
          emlrtAssign(&d_y, m2);
          e_y = nullptr;
          m3 = emlrtCreateDoubleScalar(ds - 1.0);
          emlrtAssign(&e_y, m3);
          nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
          nvtxMarkA("#b_sprintf#" MW_AT_LINE);
          emlrt_marshallIn(b_sprintf(d_y, e_y, &c_emlrtMCI),
                           "<output of sprintf>", unusedExpr);
        }
        dim = 2;
        if (cpu_dv4->size[0] != 1) {
          dim = 1;
        }
        r_outdatedOnCpu = false;
        r_outdatedOnGpu = false;
        profileLoopStart("ec_wt_fp_loop_8", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          cpu_sizeX[xpageoffset] = cpu_dv4->size[xpageoffset];
          r_outdatedOnGpu = true;
          cpu_lshift[xpageoffset] = cpu_dv4->size[xpageoffset];
          r_outdatedOnCpu = true;
        }
        profileLoopEnd();
        if (1 - dim >= 0) {
          nvtxMarkA("#computeNumIters#" MW_AT_LINE);
          mwGetLaunchParameters1D(
              computeNumIters(
                  static_cast<int32_T>(
                      (2.0 - (1.0 - static_cast<real_T>(dim))) / 2.0) -
                  1),
              &grid, &block, 2147483647U);
          if (r_outdatedOnCpu) {
            nvtxMarkA("#checkCudaError#" MW_AT_LINE);
            nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
            checkCudaError(hipMemcpy(*gpu_lshift, cpu_lshift, 8UL,
                                      hipMemcpyHostToDevice),
                           __FILE__, __LINE__);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            nvtxMarkA("#ec_wt_fp_kernel19#" MW_AT_LINE);
            ec_wt_fp_kernel19<<<grid, block>>>(
                1 - dim,
                static_cast<int32_T>((2.0 - (1.0 - static_cast<real_T>(dim))) /
                                     2.0) -
                    1,
                *gpu_lshift);
          }
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
              __FILE__, __LINE__);
        }
        nx = cpu_lshift[0];
        cpu_lshift[0] = cpu_lshift[1];
        if ((ds > 0.0) && (cpu_lshift[0] < 1)) {
          vlen = 1;
          xpageoffset = -1;
        } else {
          vlen = static_cast<int32_T>(ds);
          xpageoffset = cpu_lshift[0] - 1;
        }
        if (r_outdatedOnGpu) {
          nvtxMarkA("#checkCudaError#" MW_AT_LINE);
          nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
          checkCudaError(
              hipMemcpy(*gpu_sizeX, cpu_sizeX, 8UL, hipMemcpyHostToDevice),
              __FILE__, __LINE__);
        }
        nvtxMarkA("#ec_wt_fp_kernel20#" MW_AT_LINE);
        ec_wt_fp_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            vlen, xpageoffset, dim, *gpu_sizeX);
        dim = b_cpu_dv4->size[0] * b_cpu_dv4->size[1];
        nvtxMarkA("#div_s32#" MW_AT_LINE);
        b_cpu_dv4->size[0] = div_s32(xpageoffset, vlen) + 1;
        b_cpu_dv4->size[1] = nx;
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(b_cpu_dv4, dim, &n_emlrtRTEI);
        xpageoffset /= vlen;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1, xpageoffset), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(b_cpu_dv4, &b_gpu_dv4, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel21#" MW_AT_LINE);
          ec_wt_fp_kernel21<<<grid, block>>>(gpu_dv4, cpu_lshift[0], vlen,
                                             xpageoffset, nx - 1, b_gpu_dv4,
                                             b_cpu_dv4->size[0U]);
        }
        nvtxMarkA("#checkCudaError#" MW_AT_LINE);
        nvtxMarkA("#hipMemcpy#" MW_AT_LINE);
        checkCudaError(
            hipMemcpy(cpu_sizeX, *gpu_sizeX, 8UL, hipMemcpyDeviceToHost),
            __FILE__, __LINE__);
        dim = cpu_dv4->size[0] * cpu_dv4->size[1];
        cpu_dv4->size[0] = cpu_sizeX[0];
        cpu_dv4->size[1] = cpu_sizeX[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_dv4, dim, &p_emlrtRTEI);
        xpageoffset = cpu_sizeX[0] * cpu_sizeX[1] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_dv4, &gpu_dv4, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel22#" MW_AT_LINE);
          ec_wt_fp_kernel22<<<grid, block>>>(b_gpu_dv4, xpageoffset, gpu_dv4);
        }
      }
      c_y[0] = cpu_y->size[0];
      c_y[2] = cpu_y->size[2];
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(c_y[2] - 1, c_y[0] - 1), &grid,
                              &block, 2147483647U);
      if (y_needsGpuEnsureCapacity) {
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_y, &c_gpu_y, true);
      }
      y_needsGpuEnsureCapacity = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel23#" MW_AT_LINE);
        ec_wt_fp_kernel23<<<grid, block>>>(gpu_dv4, c_y[0], ch, c_y[0] - 1,
                                           c_y[2] - 1, c_gpu_y, cpu_y->size[0U],
                                           cpu_y->size[1U]);
      }
      r_outdatedOnCpu = true;
    }
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_dv4);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_dv);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r2);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_r);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_b);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&f_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&e_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfsnorm);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv4);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&c_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfs);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_xc);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xc);
  nvtxMarkA("#emxFreeStruct_cwtfilterbank#" MW_AT_LINE);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (r_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_real_T(cpu_y, &c_gpu_y);
  }
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xc);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_xc);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfs);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&f_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&e_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv4);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfsnorm);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_b);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r1);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r2);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_dv);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_dv4);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_sizeX), __FILE__, __LINE__);
  nvtxMarkA("#checkCudaError#" MW_AT_LINE);
  nvtxMarkA("#mwCudaFree#" MW_AT_LINE);
  checkCudaError(mwCudaFree(*gpu_lshift), __FILE__, __LINE__);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_wt_fp.cu)

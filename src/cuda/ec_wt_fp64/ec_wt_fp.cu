#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wt_fp.cu
//
// Code generation for function 'ec_wt_fp'
//

// Include files
#include "ec_wt_fp.h"
#include "cwtfilterbank.h"
#include "ec_wt_fp_data.h"
#include "ec_wt_fp_emxutil.h"
#include "ec_wt_fp_mexutil.h"
#include "ec_wt_fp_types.h"
#include "resample.h"
#include "rt_nonfinite.h"
#include "scaleSpectrum.h"
#include "wt.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtMCInfo emlrtMCI{
    27,                                                            // lineNo
    5,                                                             // colNo
    "error",                                                       // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/lang/error.m" // pName
};

static emlrtRTEInfo emlrtRTEI{
    28,                                                     // lineNo
    27,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    36,                                                     // lineNo
    1,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    48,                                                     // lineNo
    23,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    69,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    230,             // lineNo
    1,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    81,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    75,                                                     // lineNo
    14,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    69,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    78,                                                     // lineNo
    14,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    48,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    86,                                                     // lineNo
    19,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    78,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    75,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    287,             // lineNo
    33,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    181,                     // lineNo
    24,                      // colNo
    "combineVectorElements", // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/private/"
    "combineVectorElements.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    287,             // lineNo
    25,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    72,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    72,                                                     // lineNo
    5,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo v_emlrtRTEI{
    34,                                                     // lineNo
    1,                                                      // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo w_emlrtRTEI{
    48,                                                     // lineNo
    13,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    75,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    78,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

static emlrtRTEInfo bb_emlrtRTEI{
    267,             // lineNo
    36,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo cb_emlrtRTEI{
    282,             // lineNo
    9,               // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo db_emlrtRTEI{
    287,             // lineNo
    29,              // colNo
    "scaleSpectrum", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "scaleSpectrum.m" // pName
};

static emlrtRTEInfo eb_emlrtRTEI{
    81,                                                     // lineNo
    10,                                                     // colNo
    "ec_wt_fp",                                             // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/signal/ec_wt_fp.m" // pName
};

// Function Declarations
static void b_error(const mxArray *m, emlrtMCInfo *location);

static __global__ void ec_wt_fp_kernel1(const int32_T ch,
                                        const emxArray_real_T x,
                                        const int32_T b_x, emxArray_real_T xc,
                                        int32_T x_dim0);

static __global__ void ec_wt_fp_kernel10(const int32_T sz, emxArray_real_T b);

static __global__ void ec_wt_fp_kernel11(const emxArray_real_T b,
                                         const real_T y, const int32_T b_b,
                                         emxArray_real_T dv8);

static __global__ void ec_wt_fp_kernel12(const emxArray_real_T dv8,
                                         const int32_T b_dv8,
                                         emxArray_creal_T yc);

static __global__ void ec_wt_fp_kernel13(const int32_T vlen,
                                         const emxArray_real_T x,
                                         const int32_T b, emxArray_real_T b_b,
                                         int32_T x_dim0);

static __global__ void ec_wt_fp_kernel14(const emxArray_creal_T r2,
                                         const int32_T b_r2, const int32_T c_r2,
                                         emxArray_creal_T x, int32_T x_dim0,
                                         int32_T r2_dim0);

static __global__ void ec_wt_fp_kernel15(const emxArray_creal_T x,
                                         const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel16(const emxArray_real_T y,
                                         const int32_T b, emxArray_real_T b_y);

static __global__ void ec_wt_fp_kernel17(const emxArray_real_T y,
                                         const int32_T b_y,
                                         emxArray_creal_T yc);

static __global__ void ec_wt_fp_kernel18(const emxArray_creal_T r3,
                                         const int32_T b_r3, const int32_T c_r3,
                                         emxArray_creal_T x, int32_T x_dim0,
                                         int32_T r3_dim0);

static __global__ void ec_wt_fp_kernel19(const emxArray_creal_T x,
                                         const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel2(const emxArray_real_T r,
                                        const int32_T b_r,
                                        emxArray_real_T b_dv);

static __global__ void ec_wt_fp_kernel20(const emxArray_real_T y,
                                         const int32_T b_y,
                                         emxArray_creal_T yc);

static __global__ void ec_wt_fp_kernel21(const emxArray_creal_T r1,
                                         const int32_T b_r1, const int32_T c_r1,
                                         emxArray_creal_T yc, int32_T yc_dim0,
                                         int32_T r1_dim0);

static __global__ void ec_wt_fp_kernel3(const emxArray_real_T b_dv,
                                        const int32_T c_dv,
                                        emxArray_creal_T yc);

static __global__ void ec_wt_fp_kernel4(const emxArray_creal_T cfs,
                                        const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel5(const emxArray_real_T y,
                                        const int32_T b, emxArray_real_T x);

static __global__ void ec_wt_fp_kernel6(const real_T y,
                                        const emxArray_creal_T cfs,
                                        const int32_T nx, const int32_T cfsnorm,
                                        const int32_T vlen,
                                        emxArray_creal_T b_cfsnorm,
                                        int32_T cfs_dim0, int32_T cfsnorm_dim0);

static __global__ void ec_wt_fp_kernel7(const emxArray_creal_T cfsnorm,
                                        const int32_T b,
                                        const int32_T b_cfsnorm,
                                        emxArray_creal_T x, int32_T x_dim0,
                                        int32_T cfsnorm_dim0);

static __global__ void ec_wt_fp_kernel8(const emxArray_creal_T x,
                                        const int32_T b, emxArray_real_T y);

static __global__ void ec_wt_fp_kernel9(const emxArray_real_T y,
                                        const int32_T b, emxArray_real_T x);

// Function Definitions
static void b_error(const mxArray *m, emlrtMCInfo *location)
{
  const mxArray *pArray;
  nvtxRangePushA("#fcn#b_error#" MW_AT_LOCATION);
  pArray = m;
  emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 0, nullptr, 1, &pArray, "error",
                        true, location);
  nvtxRangePop();
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel1(
    const int32_T ch, const emxArray_real_T x, const int32_T b_x,
    emxArray_real_T xc, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    xc.data[xpageoffset] = x.data[xpageoffset + x_dim0 * ch];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void ec_wt_fp_kernel10(const int32_T sz,
                                                              emxArray_real_T b)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(sz);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b.data[xpageoffset] = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel11(
    const emxArray_real_T b, const real_T y, const int32_T b_b,
    emxArray_real_T dv8)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    dv8.data[xpageoffset] = y * b.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel12(
    const emxArray_real_T dv8, const int32_T b_dv8, emxArray_creal_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_dv8);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    yc.data[xpageoffset].re = dv8.data[xpageoffset];
    yc.data[xpageoffset].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel13(
    const int32_T vlen, const emxArray_real_T x, const int32_T b,
    emxArray_real_T b_b, int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx);
    xpageoffset = bcoef * x_dim0;
    b_b.data[bcoef] = x.data[xpageoffset];
    for (int32_T k{0}; k <= vlen - 2; k++) {
      b_b.data[bcoef] += x.data[(xpageoffset + k) + 1];
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel14(
    const emxArray_creal_T r2, const int32_T b_r2, const int32_T c_r2,
    emxArray_creal_T x, int32_T x_dim0, int32_T r2_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r2) + 1UL) *
                (static_cast<uint64_T>(b_r2) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r2) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b_r2) + 1UL));
    x.data[bcoef + x_dim0 * xpageoffset].re =
        r2.data[xpageoffset + r2_dim0 * bcoef].re;
    x.data[bcoef + x_dim0 * xpageoffset].im =
        -r2.data[xpageoffset + r2_dim0 * bcoef].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel15(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel16(
    const emxArray_real_T y, const int32_T b, emxArray_real_T b_y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    b_y.data[k] = y.data[k] * y.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel17(
    const emxArray_real_T y, const int32_T b_y, emxArray_creal_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    yc.data[xpageoffset].re = y.data[xpageoffset];
    yc.data[xpageoffset].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel18(
    const emxArray_creal_T r3, const int32_T b_r3, const int32_T c_r3,
    emxArray_creal_T x, int32_T x_dim0, int32_T r3_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r3) + 1UL) *
                (static_cast<uint64_T>(b_r3) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r3) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b_r3) + 1UL));
    x.data[bcoef + x_dim0 * xpageoffset].re =
        r3.data[xpageoffset + r3_dim0 * bcoef].re;
    x.data[bcoef + x_dim0 * xpageoffset].im =
        -r3.data[xpageoffset + r3_dim0 * bcoef].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel19(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel2(
    const emxArray_real_T r, const int32_T b_r, emxArray_real_T b_dv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    b_dv.data[xpageoffset] = r.data[xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel20(
    const emxArray_real_T y, const int32_T b_y, emxArray_creal_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    yc.data[xpageoffset].re = y.data[xpageoffset];
    yc.data[xpageoffset].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel21(
    const emxArray_creal_T r1, const int32_T b_r1, const int32_T c_r1,
    emxArray_creal_T yc, int32_T yc_dim0, int32_T r1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_r1) + 1UL) *
                (static_cast<uint64_T>(b_r1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_r1) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b_r1) + 1UL));
    yc.data[bcoef + yc_dim0 * xpageoffset].re =
        r1.data[xpageoffset + r1_dim0 * bcoef].re;
    yc.data[bcoef + yc_dim0 * xpageoffset].im =
        -r1.data[xpageoffset + r1_dim0 * bcoef].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel3(
    const emxArray_real_T b_dv, const int32_T c_dv, emxArray_creal_T yc)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_dv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T xpageoffset;
    xpageoffset = static_cast<int32_T>(idx);
    yc.data[xpageoffset].re = b_dv.data[xpageoffset];
    yc.data[xpageoffset].im = 0.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel4(
    const emxArray_creal_T cfs, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(cfs.data[k].re, cfs.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel5(
    const emxArray_real_T y, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x.data[k] = y.data[k] * y.data[k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel6(
    const real_T y, const emxArray_creal_T cfs, const int32_T nx,
    const int32_T cfsnorm, const int32_T vlen, emxArray_creal_T b_cfsnorm,
    int32_T cfs_dim0, int32_T cfsnorm_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(vlen) + 1UL) *
                (static_cast<uint64_T>(cfsnorm) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_k;
    int32_T bcoef;
    int32_T k;
    int32_T xpageoffset;
    b_k = static_cast<int32_T>(idx % (static_cast<uint64_T>(cfsnorm) + 1UL));
    k = static_cast<int32_T>((idx - static_cast<uint64_T>(b_k)) /
                             (static_cast<uint64_T>(cfsnorm) + 1UL));
    xpageoffset = nx * k + 1;
    bcoef = static_cast<int32_T>(cfs_dim0 != 1);
    b_cfsnorm.data[b_k + cfsnorm_dim0 * k].re =
        y * cfs.data[bcoef * b_k + cfs_dim0 * (xpageoffset - 1)].re;
    b_cfsnorm.data[b_k + cfsnorm_dim0 * k].im =
        y * cfs.data[bcoef * b_k + cfs_dim0 * (xpageoffset - 1)].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel7(
    const emxArray_creal_T cfsnorm, const int32_T b, const int32_T b_cfsnorm,
    emxArray_creal_T x, int32_T x_dim0, int32_T cfsnorm_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(b_cfsnorm) + 1UL) *
                (static_cast<uint64_T>(b) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T bcoef;
    int32_T xpageoffset;
    bcoef = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    xpageoffset = static_cast<int32_T>((idx - static_cast<uint64_T>(bcoef)) /
                                       (static_cast<uint64_T>(b) + 1UL));
    x.data[bcoef + x_dim0 * xpageoffset] =
        cfsnorm.data[bcoef + cfsnorm_dim0 * xpageoffset];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel8(
    const emxArray_creal_T x, const int32_T b, emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = hypot(x.data[k].re, x.data[k].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wt_fp_kernel9(
    const emxArray_real_T y, const int32_T b, emxArray_real_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    x.data[k] = y.data[k] * y.data[k];
  }
}

//
// function [y,freqs] = ec_wt_fp(x,fs,fLims,fVoices,ds,yType)
void ec_wt_fp(const emxArray_real_T *cpu_x, real_T fs, const real_T fLims[2],
              real_T fVoices, real_T ds[2], uint8_T yType,
              emxArray_cell_wrap_0 *y, emxArray_real_T *freqs)
{
  static const int32_T iv[2]{1, 48};
  static const char_T varargin_1[48]{
      '[', 'e', 'c', '_', 'w', 't', '_', 'f', 'p', ']', ' ', 'y',
      'T', 'y', 'p', 'e', ' ', 'm', 'u', 's', 't', ' ', 'b', 'e',
      ' ', 'i', 'n', 't', 'e', 'g', 'e', 'r', ' ', 'b', 'e', 't',
      'w', 'e', 'e', 'n', ' ', '0', ' ', 'a', 'n', 'd', ' ', '4'};
  coder::cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_yc;
  emxArray_creal_T c_gpu_x;
  emxArray_creal_T e_gpu_x;
  emxArray_creal_T f_gpu_x;
  emxArray_creal_T gpu_cfs;
  emxArray_creal_T gpu_cfsnorm;
  emxArray_creal_T gpu_r1;
  emxArray_creal_T gpu_r2;
  emxArray_creal_T gpu_r3;
  emxArray_creal_T gpu_yc;
  emxArray_creal_T *b_cpu_x;
  emxArray_creal_T *b_cpu_yc;
  emxArray_creal_T *c_cpu_x;
  emxArray_creal_T *cpu_cfs;
  emxArray_creal_T *cpu_cfsnorm;
  emxArray_creal_T *cpu_r1;
  emxArray_creal_T *cpu_r2;
  emxArray_creal_T *cpu_r3;
  emxArray_creal_T *cpu_yc;
  emxArray_creal_T *e_cpu_x;
  emxArray_real_T b_gpu_x;
  emxArray_real_T b_gpu_y;
  emxArray_real_T c_gpu_y;
  emxArray_real_T d_gpu_x;
  emxArray_real_T d_gpu_y;
  emxArray_real_T e_gpu_y;
  emxArray_real_T gpu_b;
  emxArray_real_T gpu_dv;
  emxArray_real_T gpu_dv8;
  emxArray_real_T gpu_r;
  emxArray_real_T gpu_x;
  emxArray_real_T gpu_xc;
  emxArray_real_T gpu_y;
  emxArray_real_T *Scales;
  emxArray_real_T *b_cpu_y;
  emxArray_real_T *c_cpu_y;
  emxArray_real_T *cpu_b;
  emxArray_real_T *cpu_dv;
  emxArray_real_T *cpu_dv8;
  emxArray_real_T *cpu_r;
  emxArray_real_T *cpu_xc;
  emxArray_real_T *cpu_y;
  emxArray_real_T *d_cpu_x;
  emxArray_real_T *d_cpu_y;
  emxArray_real_T *e_cpu_y;
  emxArray_real_T *f_cpu_x;
  const mxArray *b_y;
  const mxArray *m;
  int32_T i;
  int32_T xpageoffset;
  boolean_T r_outdatedOnCpu;
  boolean_T r_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T yc_outdatedOnGpu;
  nvtxMarkA("#entryPointStart#");
  nvtxRangePushA("#entryPointFcn#ec_wt_fp#" MW_AT_LOCATION "#M100,100");
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&b_gpu_yc);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r3);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r2);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_r1);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv8);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_b);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfsnorm);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&f_gpu_x);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_cfs);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&e_gpu_x);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&e_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_xc);
  nvtxMarkA("#gpuEmxReset_creal_T#" MW_AT_LINE);
  gpuEmxReset_creal_T(&gpu_yc);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_x);
  r_outdatedOnCpu = false;
  r_outdatedOnGpu = false;
  x_outdatedOnGpu = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInitStruct_cwtfilterbank#" MW_AT_LINE);
  emxInitStruct_cwtfilterbank(&fb, &v_emlrtRTEI, true);
  //  electroCUDA - run Continuous WAvelet Transform
  //    Intended to be compiled into a CUDA mex binary
  //    Kevin Tan, 2024 (github.com/kevmtan/electroCUDA)
  //
  //  OUTPUTS:
  //    y = transformed data
  //    freqs = CWT frequencies
  // 'ec_wt_fp:10' x (:,:){mustBeFloat}
  //  Input data
  // 'ec_wt_fp:11' fs (1,1) double
  //  Sampling rate
  // 'ec_wt_fp:12' fLims (1,2) double
  //  Frequency limits
  // 'ec_wt_fp:13' fVoices (1,1) double = 10
  //  Voices per octave
  // 'ec_wt_fp:14' ds (1,2) double = [0 0]
  //  Frequency limits
  //  Output data type [0=complex|1=magnitude|2=power|3=avgPower|4=avgDensity]
  // 'ec_wt_fp:16' yType (1,1) uint8 = uint8(0)
  // 'ec_wt_fp:18' if isempty(yType) || yType>4
  if (yType > 4) {
    // 'ec_wt_fp:19' error('[ec_wt_fp] yType must be integer between 0 and 4');
    b_y = nullptr;
    m = emlrtCreateCharArray(2, &iv[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 48, m, &varargin_1[0]);
    emlrtAssign(&b_y, m);
    nvtxMarkA("#b_error#" MW_AT_LINE);
    b_error(b_y, &emlrtMCI);
  }
  //  Prep
  // 'ec_wt_fp:23' coder.gpu.kernelfun;
  //  Trigger CUDA kernel creation
  //  Make vars
  // 'ec_wt_fp:26' nFrames = height(x);
  // 'ec_wt_fp:27' nChs = width(x);
  // 'ec_wt_fp:28' y = coder.nullcopy(cell(1,nChs));
  xpageoffset = y->size[0] * y->size[1];
  y->size[0] = 1;
  y->size[1] = cpu_x->size[1];
  nvtxMarkA("#emxEnsureCapacity_cell_wrap_0#" MW_AT_LINE);
  emxEnsureCapacity_cell_wrap_0(y, xpageoffset, &emlrtRTEI);
  //  Preallocate output
  // 'ec_wt_fp:29' if ds(1)<=0 || ds(2)<=1 || ~(ds(2)>ds(1))
  if ((ds[0] <= 0.0) || (ds[1] <= 1.0) || (!(ds[1] > ds[0]))) {
    // 'ec_wt_fp:30' ds = [0 0];
    profileLoopStart("ec_wt_fp_loop_0", __LINE__, 1 + 1, "");
    for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
      ds[xpageoffset] = 0.0;
    }
    profileLoopEnd();
  }
  //  Make CWT filter
  // 'ec_wt_fp:34' fb = cwtfilterbank(Wavelet="Morse",SignalLength=nFrames,...
  // 'ec_wt_fp:35'
  // SamplingFrequency=fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
  nvtxMarkA("#cwtfilterbank_cwtfilterbank#" MW_AT_LINE);
  coder::cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(cpu_x->size[0]),
                                     fs, fLims, fVoices);
  // 'ec_wt_fp:36' freqs = centerFrequencies(fb);
  xpageoffset = freqs->size[0];
  freqs->size[0] = fb.WaveletCenterFrequencies->size[0];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(freqs, xpageoffset, &b_emlrtRTEI);
  profileLoopStart("ec_wt_fp_loop_1", __LINE__,
                   (fb.WaveletCenterFrequencies->size[0] - 1) + 1, "");
  for (xpageoffset = 0; xpageoffset < fb.WaveletCenterFrequencies->size[0];
       xpageoffset++) {
    freqs->data[xpageoffset] = fb.WaveletCenterFrequencies->data[xpageoffset];
  }
  profileLoopEnd();
  //  Find initial CWT output size per channel
  // 'ec_wt_fp:39' if yType==4 || yType==3
  //  Processing loop across channels
  // 'ec_wt_fp:47' for ch = 1:nChs
  i = cpu_x->size[1];
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_yc, 2, &w_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_xc, 1, &c_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv, 1, &h_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_y, 2, &x_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&Scales, 2, &e_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&b_cpu_y, 2, &y_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_x, 2, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfs, 2, &ab_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&c_cpu_x, 2, &i_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&c_cpu_y, 2, &bb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_y, 2, &x_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&d_cpu_x, 2, &bb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_cfsnorm, 2, &cb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&e_cpu_x, 2, &q_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&e_cpu_y, 2, &db_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&f_cpu_x, 2, &db_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_b, 2, &s_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_dv8, 1, &u_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_r, 2, &d_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r1, 2, &eb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r2, 2, &g_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&cpu_r3, 2, &i_emlrtRTEI, true);
  nvtxMarkA("#emxInit_creal_T#" MW_AT_LINE);
  emxInit_creal_T(&b_cpu_yc, 2, &l_emlrtRTEI, true);
  profileLoopStart("ec_wt_fp_loop_2", __LINE__, (i - 1) + 1, "");
  for (int32_T ch{0}; ch < i; ch++) {
    int32_T nx;
    boolean_T validLaunchParams;
    // 'ec_wt_fp:48' y{ch} = wt_lfn(fb,x(:,ch),yType,ds,ycSz);
    xpageoffset = cpu_xc->size[0];
    cpu_xc->size[0] = cpu_x->size[0];
    nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
    emxEnsureCapacity_real_T(cpu_xc, xpageoffset, &c_emlrtRTEI);
    xpageoffset = cpu_x->size[0] - 1;
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                            2147483647U);
    if (x_needsGpuEnsureCapacity) {
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real_T(cpu_xc, &gpu_xc, true);
    if (x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#ec_wt_fp_kernel1#" MW_AT_LINE);
      ec_wt_fp_kernel1<<<grid, block>>>(ch, gpu_x, xpageoffset, gpu_xc,
                                        cpu_x->size[0U]);
    }
    //  Within-channel function %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    //  Preallocate channel output
    // 'ec_wt_fp:58' if yType==4 || yType==3
    //  Run WT
    // 'ec_wt_fp:67' if yType==4
    if (yType == 4) {
      //  Scale-averaged density
      // 'ec_wt_fp:69' yc = scaleSpectrum(fb,xc,SpectrumType="density")';
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
      nvtxMarkA("#cwtfilterbank_scaleSpectrum#" MW_AT_LINE);
      coder::cwtfilterbank_scaleSpectrum(&fb, cpu_xc, cpu_r, &r_outdatedOnCpu,
                                         &gpu_r, &r_outdatedOnGpu);
      xpageoffset = cpu_dv->size[0];
      cpu_dv->size[0] = cpu_r->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_dv, xpageoffset, &d_emlrtRTEI);
      xpageoffset = cpu_r->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_r, &gpu_r, !r_outdatedOnGpu);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_dv, &gpu_dv, true);
      if (r_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, cpu_r);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel2#" MW_AT_LINE);
        ec_wt_fp_kernel2<<<grid, block>>>(gpu_r, xpageoffset, gpu_dv);
      }
      xpageoffset = cpu_yc->size[0] * cpu_yc->size[1];
      cpu_yc->size[0] = cpu_dv->size[0];
      cpu_yc->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_yc, xpageoffset, &h_emlrtRTEI);
      xpageoffset = cpu_dv->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_yc, &gpu_yc, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel3#" MW_AT_LINE);
        ec_wt_fp_kernel3<<<grid, block>>>(gpu_dv, xpageoffset, gpu_yc);
      }
      r_outdatedOnCpu = false;
      r_outdatedOnGpu = true;
    } else if (yType == 3) {
      real_T c_y;
      real_T numfac;
      int32_T vlen;
      uint32_T sz[2];
      uint32_T defaultSL_idx_1;
      // 'ec_wt_fp:70' elseif yType==3
      //  Scale-averaged power
      // 'ec_wt_fp:72' yc = scaleSpectrum(fb,xc)';
      defaultSL_idx_1 = static_cast<uint32_T>(fb.Scales->size[1]);
      xpageoffset = Scales->size[0] * Scales->size[1];
      Scales->size[0] = 1;
      Scales->size[1] = fb.Scales->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(Scales, xpageoffset, &e_emlrtRTEI);
      profileLoopStart("ec_wt_fp_loop_3", __LINE__,
                       (fb.Scales->size[1] - 1) + 1, "");
      for (xpageoffset = 0; xpageoffset < fb.Scales->size[1]; xpageoffset++) {
        Scales->data[xpageoffset] = fb.Scales->data[xpageoffset];
      }
      profileLoopEnd();
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
      nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
      coder::cwtfilterbank_wt(&fb, cpu_xc, cpu_cfs);
      nx = cpu_cfs->size[0] * cpu_cfs->size[1];
      profileLoopStart("ec_wt_fp_loop_4", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(cpu_cfs->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = c_cpu_y->size[0] * c_cpu_y->size[1];
      c_cpu_y->size[0] = static_cast<int32_T>(sz[0]);
      c_cpu_y->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(c_cpu_y, xpageoffset, &j_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_cfs, &gpu_cfs, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(c_cpu_y, &gpu_y, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_cfs, cpu_cfs);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel4#" MW_AT_LINE);
        ec_wt_fp_kernel4<<<grid, block>>>(gpu_cfs, nx - 1, gpu_y);
      }
      profileLoopStart("ec_wt_fp_loop_7", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(c_cpu_y->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = d_cpu_x->size[0] * d_cpu_x->size[1];
      d_cpu_x->size[0] = static_cast<int32_T>(sz[0]);
      d_cpu_x->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(d_cpu_x, xpageoffset, &m_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(
          computeNumIters(
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
          &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(d_cpu_x, &b_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel5#" MW_AT_LINE);
        ec_wt_fp_kernel5<<<grid, block>>>(
            gpu_y,
            static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
            b_gpu_x);
      }
      vlen = d_cpu_x->size[0] * d_cpu_x->size[1];
      if (d_cpu_x->size[0] * d_cpu_x->size[1] == 0) {
        c_y = 0.0;
      } else {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real_T(d_cpu_x, &b_gpu_x);
        c_y = d_cpu_x->data[0];
        profileLoopStart("ec_wt_fp_loop_11", __LINE__, (vlen - 2) + 1, "");
        for (nx = 0; nx <= vlen - 2; nx++) {
          c_y += d_cpu_x->data[nx + 1];
        }
        profileLoopEnd();
      }
      numfac = fb.sigvar;
      c_y = std::sqrt(numfac /
                      (1.0 / static_cast<real_T>(Scales->size[1]) * c_y));
      xpageoffset = cpu_cfsnorm->size[0] * cpu_cfsnorm->size[1];
      cpu_cfsnorm->size[0] = cpu_cfs->size[0];
      cpu_cfsnorm->size[1] = cpu_cfs->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_cfsnorm, xpageoffset, &p_emlrtRTEI);
      r_outdatedOnCpu = true;
      if ((cpu_cfs->size[0] != 0) && (cpu_cfs->size[1] != 0)) {
        nx = (cpu_cfs->size[1] != 1);
        vlen = cpu_cfs->size[1] - 1;
        xpageoffset = cpu_cfsnorm->size[0] - 1;
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(vlen, xpageoffset), &grid,
                                &block, 2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_cfsnorm, &gpu_cfsnorm, true);
        r_outdatedOnCpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel6#" MW_AT_LINE);
          ec_wt_fp_kernel6<<<grid, block>>>(c_y, gpu_cfs, nx, xpageoffset, vlen,
                                            gpu_cfsnorm, cpu_cfs->size[0U],
                                            cpu_cfsnorm->size[0U]);
        }
      }
      c_y = 1.0 / static_cast<real_T>(Scales->size[1]);
      xpageoffset = e_cpu_x->size[0] * e_cpu_x->size[1];
      e_cpu_x->size[0] = static_cast<int32_T>(defaultSL_idx_1);
      e_cpu_x->size[1] = cpu_cfsnorm->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(e_cpu_x, xpageoffset, &q_emlrtRTEI);
      xpageoffset = cpu_cfsnorm->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(
          computeNumIters(xpageoffset,
                          static_cast<int32_T>(defaultSL_idx_1) - 1),
          &grid, &block, 2147483647U);
      if (r_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_creal_T(cpu_cfsnorm, &gpu_cfsnorm, true);
      }
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(e_cpu_x, &c_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel7#" MW_AT_LINE);
        ec_wt_fp_kernel7<<<grid, block>>>(
            gpu_cfsnorm, static_cast<int32_T>(defaultSL_idx_1) - 1, xpageoffset,
            c_gpu_x, e_cpu_x->size[0U], cpu_cfsnorm->size[0U]);
      }
      nx = static_cast<int32_T>(defaultSL_idx_1) * cpu_cfsnorm->size[1];
      xpageoffset = e_cpu_y->size[0] * e_cpu_y->size[1];
      e_cpu_y->size[0] = static_cast<int32_T>(defaultSL_idx_1);
      e_cpu_y->size[1] = cpu_cfsnorm->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(e_cpu_y, xpageoffset, &j_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(e_cpu_y, &b_gpu_y, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel8#" MW_AT_LINE);
        ec_wt_fp_kernel8<<<grid, block>>>(c_gpu_x, nx - 1, b_gpu_y);
      }
      profileLoopStart("ec_wt_fp_loop_12", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(e_cpu_y->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = f_cpu_x->size[0] * f_cpu_x->size[1];
      f_cpu_x->size[0] = static_cast<int32_T>(sz[0]);
      f_cpu_x->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(f_cpu_x, xpageoffset, &m_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(
          computeNumIters(
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
          &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(f_cpu_x, &d_gpu_x, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel9#" MW_AT_LINE);
        ec_wt_fp_kernel9<<<grid, block>>>(
            b_gpu_y,
            static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
            d_gpu_x);
      }
      vlen = f_cpu_x->size[0];
      if ((f_cpu_x->size[0] == 0) || (f_cpu_x->size[1] == 0)) {
        profileLoopStart("ec_wt_fp_loop_13", __LINE__, 1 + 1, "");
        for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
          sz[xpageoffset] = static_cast<uint32_T>(f_cpu_x->size[xpageoffset]);
        }
        profileLoopEnd();
        xpageoffset = cpu_b->size[0] * cpu_b->size[1];
        cpu_b->size[0] = 1;
        cpu_b->size[1] = static_cast<int32_T>(sz[1]);
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_b, xpageoffset, &s_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(
            computeNumIters(static_cast<int32_T>(sz[1]) - 1), &grid, &block,
            2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel10#" MW_AT_LINE);
          ec_wt_fp_kernel10<<<grid, block>>>(static_cast<int32_T>(sz[1]) - 1,
                                             gpu_b);
        }
      } else {
        nx = f_cpu_x->size[1];
        xpageoffset = cpu_b->size[0] * cpu_b->size[1];
        cpu_b->size[0] = 1;
        cpu_b->size[1] = f_cpu_x->size[1];
        nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
        emxEnsureCapacity_real_T(cpu_b, xpageoffset, &r_emlrtRTEI);
        nvtxMarkA("#computeNumIters#" MW_AT_LINE);
        mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                                2147483647U);
        nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
        gpuEmxEnsureCapacity_real_T(cpu_b, &gpu_b, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          nvtxMarkA("#ec_wt_fp_kernel13#" MW_AT_LINE);
          ec_wt_fp_kernel13<<<grid, block>>>(vlen, d_gpu_x, nx - 1, gpu_b,
                                             f_cpu_x->size[0U]);
        }
      }
      xpageoffset = cpu_dv8->size[0];
      cpu_dv8->size[0] = cpu_b->size[1];
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_dv8, xpageoffset, &t_emlrtRTEI);
      xpageoffset = cpu_b->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_dv8, &gpu_dv8, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel11#" MW_AT_LINE);
        ec_wt_fp_kernel11<<<grid, block>>>(gpu_b, c_y, xpageoffset, gpu_dv8);
      }
      xpageoffset = cpu_yc->size[0] * cpu_yc->size[1];
      cpu_yc->size[0] = cpu_dv8->size[0];
      cpu_yc->size[1] = 1;
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_yc, xpageoffset, &u_emlrtRTEI);
      xpageoffset = cpu_dv8->size[0] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_yc, &gpu_yc, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel12#" MW_AT_LINE);
        ec_wt_fp_kernel12<<<grid, block>>>(gpu_dv8, xpageoffset, gpu_yc);
      }
      r_outdatedOnCpu = false;
      r_outdatedOnGpu = true;
    } else if (yType == 2) {
      int32_T vlen;
      uint32_T sz[2];
      // 'ec_wt_fp:73' elseif yType==2
      //  CWT power
      // 'ec_wt_fp:75' yc = abs(wt(fb,xc)').^2;
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
      nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
      coder::cwtfilterbank_wt(&fb, cpu_xc, cpu_r2);
      xpageoffset = b_cpu_x->size[0] * b_cpu_x->size[1];
      b_cpu_x->size[0] = cpu_r2->size[1];
      b_cpu_x->size[1] = cpu_r2->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(b_cpu_x, xpageoffset, &g_emlrtRTEI);
      xpageoffset = cpu_r2->size[0] - 1;
      vlen = cpu_r2->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset, vlen), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_r2, &gpu_r2, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(b_cpu_x, &e_gpu_x, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r2, cpu_r2);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel14#" MW_AT_LINE);
        ec_wt_fp_kernel14<<<grid, block>>>(gpu_r2, vlen, xpageoffset, e_gpu_x,
                                           b_cpu_x->size[0U], cpu_r2->size[0U]);
      }
      nx = b_cpu_x->size[0] * b_cpu_x->size[1];
      profileLoopStart("ec_wt_fp_loop_5", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(b_cpu_x->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = cpu_y->size[0] * cpu_y->size[1];
      cpu_y->size[0] = static_cast<int32_T>(sz[0]);
      cpu_y->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(cpu_y, xpageoffset, &j_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(cpu_y, &c_gpu_y, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel15#" MW_AT_LINE);
        ec_wt_fp_kernel15<<<grid, block>>>(e_gpu_x, nx - 1, c_gpu_y);
      }
      profileLoopStart("ec_wt_fp_loop_8", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(cpu_y->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = d_cpu_y->size[0] * d_cpu_y->size[1];
      d_cpu_y->size[0] = static_cast<int32_T>(sz[0]);
      d_cpu_y->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(d_cpu_y, xpageoffset, &m_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(
          computeNumIters(
              static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1),
          &grid, &block, 2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(d_cpu_y, &d_gpu_y, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel16#" MW_AT_LINE);
        ec_wt_fp_kernel16<<<grid, block>>>(
            c_gpu_y,
            static_cast<int32_T>(sz[0]) * static_cast<int32_T>(sz[1]) - 1,
            d_gpu_y);
      }
      xpageoffset = cpu_yc->size[0] * cpu_yc->size[1];
      cpu_yc->size[0] = d_cpu_y->size[0];
      cpu_yc->size[1] = d_cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_yc, xpageoffset, &o_emlrtRTEI);
      xpageoffset = d_cpu_y->size[0] * d_cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_yc, &gpu_yc, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel17#" MW_AT_LINE);
        ec_wt_fp_kernel17<<<grid, block>>>(d_gpu_y, xpageoffset, gpu_yc);
      }
      r_outdatedOnCpu = false;
      r_outdatedOnGpu = true;
    } else if (yType == 1) {
      int32_T vlen;
      uint32_T sz[2];
      // 'ec_wt_fp:76' elseif yType==1
      //  CWT magnitude
      // 'ec_wt_fp:78' yc = abs(wt(fb,xc)');
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
      nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
      coder::cwtfilterbank_wt(&fb, cpu_xc, cpu_r3);
      xpageoffset = c_cpu_x->size[0] * c_cpu_x->size[1];
      c_cpu_x->size[0] = cpu_r3->size[1];
      c_cpu_x->size[1] = cpu_r3->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(c_cpu_x, xpageoffset, &i_emlrtRTEI);
      xpageoffset = cpu_r3->size[0] - 1;
      vlen = cpu_r3->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset, vlen), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_r3, &gpu_r3, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(c_cpu_x, &f_gpu_x, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r3, cpu_r3);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel18#" MW_AT_LINE);
        ec_wt_fp_kernel18<<<grid, block>>>(gpu_r3, vlen, xpageoffset, f_gpu_x,
                                           c_cpu_x->size[0U], cpu_r3->size[0U]);
      }
      nx = c_cpu_x->size[0] * c_cpu_x->size[1];
      profileLoopStart("ec_wt_fp_loop_6", __LINE__, 1 + 1, "");
      for (xpageoffset = 0; xpageoffset < 2; xpageoffset++) {
        sz[xpageoffset] = static_cast<uint32_T>(c_cpu_x->size[xpageoffset]);
      }
      profileLoopEnd();
      xpageoffset = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = static_cast<int32_T>(sz[0]);
      b_cpu_y->size[1] = static_cast<int32_T>(sz[1]);
      nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
      emxEnsureCapacity_real_T(b_cpu_y, xpageoffset, &j_emlrtRTEI);
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(nx - 1), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_real_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_real_T(b_cpu_y, &e_gpu_y, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel19#" MW_AT_LINE);
        ec_wt_fp_kernel19<<<grid, block>>>(f_gpu_x, nx - 1, e_gpu_y);
      }
      xpageoffset = cpu_yc->size[0] * cpu_yc->size[1];
      cpu_yc->size[0] = b_cpu_y->size[0];
      cpu_yc->size[1] = b_cpu_y->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_yc, xpageoffset, &n_emlrtRTEI);
      xpageoffset = b_cpu_y->size[0] * b_cpu_y->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_yc, &gpu_yc, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel20#" MW_AT_LINE);
        ec_wt_fp_kernel20<<<grid, block>>>(e_gpu_y, xpageoffset, gpu_yc);
      }
      r_outdatedOnCpu = false;
      r_outdatedOnGpu = true;
    } else {
      int32_T vlen;
      // 'ec_wt_fp:79' elseif yType==0
      //  CWT coefficients (complex-valued)
      // 'ec_wt_fp:81' yc = wt(fb,xc)';
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real_T(cpu_xc, &gpu_xc);
      nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
      coder::cwtfilterbank_wt(&fb, cpu_xc, cpu_r1);
      xpageoffset = cpu_yc->size[0] * cpu_yc->size[1];
      cpu_yc->size[0] = cpu_r1->size[1];
      cpu_yc->size[1] = cpu_r1->size[0];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(cpu_yc, xpageoffset, &f_emlrtRTEI);
      xpageoffset = cpu_r1->size[0] - 1;
      vlen = cpu_r1->size[1] - 1;
      nvtxMarkA("#computeNumIters#" MW_AT_LINE);
      mwGetLaunchParameters1D(computeNumIters(xpageoffset, vlen), &grid, &block,
                              2147483647U);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_r1, &gpu_r1, false);
      nvtxMarkA("#gpuEmxEnsureCapacity_creal_T#" MW_AT_LINE);
      gpuEmxEnsureCapacity_creal_T(cpu_yc, &gpu_yc, true);
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal_T(&gpu_r1, cpu_r1);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        nvtxMarkA("#ec_wt_fp_kernel21#" MW_AT_LINE);
        ec_wt_fp_kernel21<<<grid, block>>>(gpu_r1, vlen, xpageoffset, gpu_yc,
                                           cpu_yc->size[0U], cpu_r1->size[0U]);
      }
      r_outdatedOnCpu = false;
      r_outdatedOnGpu = true;
    }
    //  Downsample
    // 'ec_wt_fp:85' if ds(2)
    if (ds[1] != 0.0) {
      // 'ec_wt_fp:86' yc = resample(yc,ds(1),ds(2));
      yc_outdatedOnGpu = false;
      xpageoffset = b_cpu_yc->size[0] * b_cpu_yc->size[1];
      b_cpu_yc->size[0] = cpu_yc->size[0];
      b_cpu_yc->size[1] = cpu_yc->size[1];
      nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
      emxEnsureCapacity_creal_T(b_cpu_yc, xpageoffset, &l_emlrtRTEI);
      nx = cpu_yc->size[0] * cpu_yc->size[1] - 1;
      profileLoopStart("ec_wt_fp_loop_10", __LINE__, nx + 1, "");
      for (xpageoffset = 0; xpageoffset <= nx; xpageoffset++) {
        if (r_outdatedOnGpu) {
          nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
          gpuEmxMemcpyGpuToCpu_creal_T(cpu_yc, &gpu_yc);
        }
        r_outdatedOnGpu = false;
        b_cpu_yc->data[xpageoffset] = cpu_yc->data[xpageoffset];
        yc_outdatedOnGpu = true;
      }
      profileLoopEnd();
      nvtxMarkA("#resample#" MW_AT_LINE);
      coder::resample(b_cpu_yc, &b_gpu_yc, &yc_outdatedOnGpu, ds[0], ds[1],
                      cpu_yc, &r_outdatedOnGpu, &gpu_yc, &r_outdatedOnCpu);
    }
    xpageoffset = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
    y->data[ch].f1->size[0] = cpu_yc->size[0];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(y->data[ch].f1, xpageoffset, &k_emlrtRTEI);
    xpageoffset = y->data[ch].f1->size[0] * y->data[ch].f1->size[1];
    y->data[ch].f1->size[1] = cpu_yc->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal_T#" MW_AT_LINE);
    emxEnsureCapacity_creal_T(y->data[ch].f1, xpageoffset, &k_emlrtRTEI);
    profileLoopStart("ec_wt_fp_loop_9", __LINE__,
                     (cpu_yc->size[0] * cpu_yc->size[1] - 1) + 1, "");
    for (xpageoffset = 0; xpageoffset < cpu_yc->size[0] * cpu_yc->size[1];
         xpageoffset++) {
      if (r_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_creal_T(cpu_yc, &gpu_yc);
      }
      r_outdatedOnGpu = false;
      y->data[ch].f1->data[xpageoffset] = cpu_yc->data[xpageoffset];
    }
    profileLoopEnd();
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_yc);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r3);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r2);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_r1);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_r);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv8);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_b);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&f_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&e_cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&e_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfsnorm);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&d_cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&c_cpu_y);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&c_cpu_x);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_cfs);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&b_cpu_x);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&b_cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&Scales);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_y);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_dv);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_xc);
  nvtxMarkA("#emxFree_creal_T#" MW_AT_LINE);
  emxFree_creal_T(&cpu_yc);
  nvtxMarkA("#emxFreeStruct_cwtfilterbank#" MW_AT_LINE);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_yc);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_xc);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&c_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&e_gpu_y);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&e_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfs);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&f_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_x);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_cfsnorm);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&c_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&d_gpu_x);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_b);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_dv8);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_r);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r1);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r2);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&gpu_r3);
  nvtxMarkA("#gpuEmxFree_creal_T#" MW_AT_LINE);
  gpuEmxFree_creal_T(&b_gpu_yc);
  nvtxRangePop();
  nvtxMarkA("#entryPointEnd#");
}

// End of code generation (ec_wt_fp.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wtcc.cu
//
// Code generation for function 'ec_wtcc'
//

// Include files
#include "ec_wtcc.h"
#include "conv2.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "firls.h"
#include "kaiser.h"
#include "mod.h"
#include "rat.h"
#include "resample.h"
#include "rt_nonfinite.h"
#include "uniformResampleKernel.h"
#include "wcoherence.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hip/hip_math_constants.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo emlrtRTEI{
    1,                                                   // lineNo
    20,                                                  // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo b_emlrtRTEI{
    19,                                                  // lineNo
    21,                                                  // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    28,                                                  // lineNo
    29,                                                  // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    28,                                                  // lineNo
    41,                                                  // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    39,                                                  // lineNo
    1,                                                   // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    39,                                                  // lineNo
    9,                                                   // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    28,                                                  // lineNo
    12,                                                  // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    28,                                                  // lineNo
    6,                                                   // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    43,                                                  // lineNo
    5,                                                   // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    44,                                                  // lineNo
    19,                                                  // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo x_emlrtRTEI{
    35,                                                  // lineNo
    14,                                                  // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

static emlrtRTEInfo y_emlrtRTEI{
    48,                                                  // lineNo
    6,                                                   // colNo
    "ec_wtcc",                                           // fName
    "/home/kt/Gdrive/Git/electroCUDA/src/cuda/ec_wtcc.m" // pName
};

// Function Declarations
static __global__ void ec_wtcc_kernel1(const int32_T c, const int32_T vlen,
                                       const emxArray_real32_T x,
                                       const int32_T b_x, emxArray_real32_T x2,
                                       emxArray_real32_T x1, int32_T x_dim0);

static __global__ void ec_wtcc_kernel10(const emxArray_uint32_T y,
                                        const real_T nZeroBegin,
                                        const int32_T b_y,
                                        emxArray_int32_T iv1);

static __global__ void ec_wtcc_kernel11(const emxArray_real32_T opts_filter,
                                        const emxArray_int32_T iv1,
                                        const int32_T b_iv1,
                                        emxArray_real32_T h);

static __global__ void ec_wtcc_kernel12(const emxArray_real32_T opts_x,
                                        const int32_T b_opts_x,
                                        emxArray_real32_T b_y1);

static __global__ void ec_wtcc_kernel13(const emxArray_real32_T b_y1,
                                        const int32_T b, emxArray_real32_T fv);

static __global__ void ec_wtcc_kernel14(const emxArray_creal32_T yp,
                                        const int32_T b, emxArray_real32_T y);

static __global__ void ec_wtcc_kernel15(const emxArray_real32_T opts_x,
                                        const int32_T b_opts_x,
                                        emxArray_real32_T fv);

static __global__ void ec_wtcc_kernel16(const emxArray_real32_T w,
                                        const int32_T vlen, const int32_T b,
                                        emxArray_real32_T y);

static __global__ void ec_wtcc_kernel17(emxArray_real_T y);

static __global__ void
ec_wtcc_kernel18(const emxArray_real32_T y, const emxArray_real_T b_y,
                 const real_T opts_filterDelay, const int32_T ii,
                 const int32_T b, emxArray_real32_T b_y1, int32_T y1_dim0);

static __global__ void ec_wtcc_kernel19(const int32_T b, emxArray_real_T y);

static __global__ void ec_wtcc_kernel2(const emxArray_real32_T xp,
                                       const int32_T b_xp, const int32_T c_xp,
                                       emxArray_real32_T fv, int32_T fv_dim0,
                                       int32_T xp_dim0);

static __global__ void ec_wtcc_kernel20(const emxArray_real32_T b_y1,
                                        const int32_T b, emxArray_real32_T fv);

static __global__ void ec_wtcc_kernel21(const emxArray_real32_T b_y1,
                                        const int32_T c_y1, const int32_T d_y1,
                                        emxArray_real32_T fv, int32_T fv_dim0,
                                        int32_T y1_dim0);

static __global__ void ec_wtcc_kernel22(const emxArray_real32_T opts_x,
                                        const int32_T b_opts_x,
                                        emxArray_real32_T b_y1);

static __global__ void ec_wtcc_kernel23(const int32_T b, emxArray_uint32_T y);

static __global__ void ec_wtcc_kernel24(const emxArray_real32_T fv,
                                        const int32_T b_fv,
                                        emxArray_real32_T opts_x);

static __global__ void ec_wtcc_kernel25(const emxArray_real32_T fv,
                                        const int32_T b_fv, const int32_T c_fv,
                                        emxArray_real32_T opts_x,
                                        int32_T opts_x_dim0, int32_T fv_dim0);

static __global__ void ec_wtcc_kernel3(const emxArray_creal32_T yp,
                                       const int32_T b_yp, const int32_T c_yp,
                                       emxArray_creal32_T d_yp, int32_T yp_dim0,
                                       int32_T b_yp_dim0);

static __global__ void ec_wtcc_kernel4(const emxArray_real32_T fv,
                                       const int32_T b,
                                       emxArray_real32_T opts_x);

static __global__ void ec_wtcc_kernel5(const emxArray_real_T r,
                                       const int32_T b_r,
                                       emxArray_real_T b_dv1);

static __global__ void ec_wtcc_kernel6(const emxArray_real_T b_dv2,
                                       const emxArray_real_T b_dv1,
                                       const int32_T c_dv1, emxArray_real_T h1);

static __global__ void ec_wtcc_kernel7(const real_T pqmax, const real_T opts_p,
                                       const int32_T h1, emxArray_real_T b_h1);

static __global__ void ec_wtcc_kernel8(const emxArray_real_T h1,
                                       const int32_T b_h1,
                                       emxArray_real32_T opts_filter);

static __global__ void ec_wtcc_kernel9(const int32_T nZeroBegin,
                                       emxArray_real32_T h);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel1(
    const int32_T c, const int32_T vlen, const emxArray_real32_T x,
    const int32_T b_x, emxArray_real32_T x2, emxArray_real32_T x1,
    int32_T x_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    x1.data[i1] = x.data[i1 + x_dim0 * (vlen - 1)];
    x2.data[i1] = x.data[i1 + x_dim0 * (c - 1)];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel10(
    const emxArray_uint32_T y, const real_T nZeroBegin, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    iv1.data[i1] =
        static_cast<int32_T>(nZeroBegin + static_cast<real_T>(y.data[i1]));
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel11(
    const emxArray_real32_T opts_filter, const emxArray_int32_T iv1,
    const int32_T b_iv1, emxArray_real32_T h)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_iv1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h.data[iv1.data[i1] - 1] = opts_filter.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel12(
    const emxArray_real32_T opts_x, const int32_T b_opts_x,
    emxArray_real32_T b_y1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_opts_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1] = opts_x.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel13(
    const emxArray_real32_T b_y1, const int32_T b, emxArray_real32_T fv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    fv.data[i1] = b_y1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel14(
    const emxArray_creal32_T yp, const int32_T b, emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k] = atan2f(yp.data[k].im, yp.data[k].re);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel15(
    const emxArray_real32_T opts_x, const int32_T b_opts_x,
    emxArray_real32_T fv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_opts_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    fv.data[i1] = opts_x.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel16(
    const emxArray_real32_T w, const int32_T vlen, const int32_T b,
    emxArray_real32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = w.data[vlen * i1];
  }
}

static __global__ __launch_bounds__(32,
                                    1) void ec_wtcc_kernel17(emxArray_real_T y)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    y.data[0] = HIP_NAN;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel18(
    const emxArray_real32_T y, const emxArray_real_T b_y,
    const real_T opts_filterDelay, const int32_T ii, const int32_T b,
    emxArray_real32_T b_y1, int32_T y1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1 + y1_dim0 * ii] =
        y.data[static_cast<int32_T>(opts_filterDelay + b_y.data[i1]) - 1];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void ec_wtcc_kernel19(const int32_T b,
                                                             emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<real_T>(i1) + 1.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel2(
    const emxArray_real32_T xp, const int32_T b_xp, const int32_T c_xp,
    emxArray_real32_T fv, int32_T fv_dim0, int32_T xp_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_xp) + 1UL) *
                (static_cast<uint64_T>(b_xp) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_xp) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_xp) + 1UL));
    fv.data[k + fv_dim0 * i1] = xp.data[i1 + xp_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel20(
    const emxArray_real32_T b_y1, const int32_T b, emxArray_real32_T fv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    fv.data[i1] = b_y1.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel21(
    const emxArray_real32_T b_y1, const int32_T c_y1, const int32_T d_y1,
    emxArray_real32_T fv, int32_T fv_dim0, int32_T y1_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(d_y1) + 1UL) *
                (static_cast<uint64_T>(c_y1) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(c_y1) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(c_y1) + 1UL));
    fv.data[k + fv_dim0 * i1] = b_y1.data[i1 + y1_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel22(
    const emxArray_real32_T opts_x, const int32_T b_opts_x,
    emxArray_real32_T b_y1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_opts_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y1.data[i1] = opts_x.data[i1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_wtcc_kernel23(const int32_T b,
                                                     emxArray_uint32_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y.data[i1] = static_cast<uint32_T>(i1) + 1U;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel24(
    const emxArray_real32_T fv, const int32_T b_fv, emxArray_real32_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_fv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = fv.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel25(
    const emxArray_real32_T fv, const int32_T b_fv, const int32_T c_fv,
    emxArray_real32_T opts_x, int32_T opts_x_dim0, int32_T fv_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_fv) + 1UL) *
                (static_cast<uint64_T>(b_fv) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_fv) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_fv) + 1UL));
    opts_x.data[k + opts_x_dim0 * i1] = fv.data[i1 + fv_dim0 * k];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel3(
    const emxArray_creal32_T yp, const int32_T b_yp, const int32_T c_yp,
    emxArray_creal32_T d_yp, int32_T yp_dim0, int32_T b_yp_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_yp) + 1UL) *
                (static_cast<uint64_T>(b_yp) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T k;
    k = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_yp) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(k)) /
                              (static_cast<uint64_T>(b_yp) + 1UL));
    d_yp.data[k + yp_dim0 * i1].re = yp.data[i1 + b_yp_dim0 * k].re;
    d_yp.data[k + yp_dim0 * i1].im = -yp.data[i1 + b_yp_dim0 * k].im;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel4(
    const emxArray_real32_T fv, const int32_T b, emxArray_real32_T opts_x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_x.data[i1] = fv.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel5(
    const emxArray_real_T r, const int32_T b_r, emxArray_real_T b_dv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_r);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_dv1.data[i1] = r.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel6(
    const emxArray_real_T b_dv2, const emxArray_real_T b_dv1,
    const int32_T c_dv1, emxArray_real_T h1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_dv1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h1.data[i1] = b_dv1.data[i1] * b_dv2.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel7(
    const real_T pqmax, const real_T opts_p, const int32_T h1,
    emxArray_real_T b_h1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(h1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_h1.data[i1] = opts_p * b_h1.data[i1] / pqmax;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_wtcc_kernel8(
    const emxArray_real_T h1, const int32_T b_h1, emxArray_real32_T opts_filter)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_h1);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    opts_filter.data[i1] = static_cast<real32_T>(h1.data[i1]);
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_wtcc_kernel9(const int32_T nZeroBegin,
                                                    emxArray_real32_T h)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nZeroBegin);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    h.data[i1] = 0.0F;
  }
}

//
// function [xx,yy] = ec_wtcc(x,c,fs,fLims,fVoices,ds)
void ec_wtcc(const emxArray_real32_T *cpu_x, const emxArray_uint16_T *c,
             real_T fs, const real_T fLims[2], real_T fVoices, real_T ds[2],
             emxArray_cell_wrap_0 *xx, emxArray_cell_wrap_0 *yy)
{
  static const int32_T iv2[2]{1, 7};
  static const int32_T iv3[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  dim3 block;
  dim3 grid;
  emxArray_creal32_T b_gpu_yp;
  emxArray_creal32_T c_gpu_yp;
  emxArray_creal32_T gpu_yp;
  emxArray_creal32_T *b_cpu_yp;
  emxArray_creal32_T *c_cpu_yp;
  emxArray_creal32_T *cpu_yp;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T *cpu_iv1;
  emxArray_real32_T b_gpu_y;
  emxArray_real32_T b_yCol;
  emxArray_real32_T c_gpu_y;
  emxArray_real32_T gpu_fv;
  emxArray_real32_T gpu_h;
  emxArray_real32_T gpu_opts_filter;
  emxArray_real32_T gpu_opts_x;
  emxArray_real32_T gpu_w;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_x1;
  emxArray_real32_T gpu_x2;
  emxArray_real32_T gpu_xp;
  emxArray_real32_T gpu_y1;
  emxArray_real32_T *c_cpu_y;
  emxArray_real32_T *cpu_fv;
  emxArray_real32_T *cpu_h;
  emxArray_real32_T *cpu_opts_filter;
  emxArray_real32_T *cpu_opts_x;
  emxArray_real32_T *cpu_w;
  emxArray_real32_T *cpu_x1;
  emxArray_real32_T *cpu_x2;
  emxArray_real32_T *cpu_xp;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *cpu_y1;
  emxArray_real32_T *yCol;
  emxArray_real_T d_gpu_y;
  emxArray_real_T gpu_dv1;
  emxArray_real_T gpu_dv2;
  emxArray_real_T gpu_h1;
  emxArray_real_T gpu_r;
  emxArray_real_T *cpu_dv1;
  emxArray_real_T *cpu_dv2;
  emxArray_real_T *cpu_h1;
  emxArray_real_T *cpu_r;
  emxArray_real_T *d_cpu_y;
  emxArray_uint32_T gpu_y;
  emxArray_uint32_T *b_cpu_y;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T Ly;
  real_T opts_q;
  int32_T opts_x[1];
  int32_T i;
  int32_T i1;
  int32_T i3;
  int32_T i4;
  char_T unusedExpr[23];
  boolean_T b_y_needsGpuEnsureCapacity;
  boolean_T b_yp_outdatedOnCpu;
  boolean_T fv_outdatedOnCpu;
  boolean_T h_outdatedOnCpu;
  boolean_T h_outdatedOnGpu;
  boolean_T opts_isRowVectorInput;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T x_needsGpuEnsureCapacity;
  boolean_T x_outdatedOnGpu;
  boolean_T y_needsGpuEnsureCapacity;
  boolean_T yp_needsGpuEnsureCapacity;
  boolean_T yp_outdatedOnCpu;
  boolean_T yp_outdatedOnGpu;
  gpuEmxReset_creal32_T(&c_gpu_yp);
  gpuEmxReset_real_T(&gpu_r);
  gpuEmxReset_real_T(&d_gpu_y);
  gpuEmxReset_real32_T(&c_gpu_y);
  gpuEmxReset_real32_T(&gpu_w);
  gpuEmxReset_real32_T(&gpu_y1);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_uint32_T(&gpu_y);
  gpuEmxReset_real32_T(&gpu_h);
  gpuEmxReset_real_T(&gpu_h1);
  gpuEmxReset_real_T(&gpu_dv2);
  gpuEmxReset_real_T(&gpu_dv1);
  gpuEmxReset_real32_T(&gpu_opts_x);
  gpuEmxReset_real32_T(&gpu_opts_filter);
  gpuEmxReset_real32_T(&b_gpu_y);
  gpuEmxReset_creal32_T(&b_gpu_yp);
  gpuEmxReset_real32_T(&gpu_fv);
  gpuEmxReset_creal32_T(&gpu_yp);
  gpuEmxReset_real32_T(&gpu_xp);
  gpuEmxReset_real32_T(&gpu_x2);
  gpuEmxReset_real32_T(&gpu_x1);
  gpuEmxReset_real32_T(&gpu_x);
  w_outdatedOnCpu = false;
  w_outdatedOnGpu = false;
  yp_outdatedOnCpu = false;
  yp_outdatedOnGpu = false;
  x_outdatedOnGpu = true;
  b_y_needsGpuEnsureCapacity = true;
  y_needsGpuEnsureCapacity = true;
  x_needsGpuEnsureCapacity = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  Wavelet coherence & resample
  // 'ec_wtcc:4' x (:,:){mustBeFloat}
  // 'ec_wtcc:5' c (:,2) uint16
  // 'ec_wtcc:6' fs (1,1) double
  // 'ec_wtcc:7' fLims (1,2) double
  // 'ec_wtcc:8' fVoices (1,1) double = 12
  // 'ec_wtcc:9' ds (1,2) double = [1 1]
  //  Trigger kernel creation
  // 'ec_wtcc:13' coder.gpu.kernelfun;
  //  Make vars
  // 'ec_wtcc:16' nPairs = height(c);
  // 'ec_wtcc:17' tmp = cell(1,nPairs);
  i = xx->size[0] * xx->size[1];
  xx->size[0] = 1;
  xx->size[1] = c->size[0];
  emxEnsureCapacity_cell_wrap_0(xx, i, &emlrtRTEI);
  // 'ec_wtcc:18' xx = coder.nullcopy(tmp);
  //  Fix define array error
  // 'ec_wtcc:19' yy = coder.nullcopy(tmp);
  i = yy->size[0] * yy->size[1];
  yy->size[0] = 1;
  yy->size[1] = c->size[0];
  emxEnsureCapacity_cell_wrap_0(yy, i, &b_emlrtRTEI);
  // 'ec_wtcc:21' if ds(1)==0 || ds(2)==0 || ~(ds(2)>ds(1))
  if ((ds[0] == 0.0) || (ds[1] == 0.0) || (!(ds[1] > ds[0]))) {
    // 'ec_wtcc:22' ds(1) = 0;
    ds[0] = 0.0;
    // 'ec_wtcc:23' ds(2) = 0;
    ds[1] = 0.0;
  }
  //  Loop across pairs
  // 'ec_wtcc:27' for p = 1:nPairs
  i1 = c->size[0];
  emxInit_real32_T(&cpu_x1, 1, &c_emlrtRTEI, true);
  emxInit_real32_T(&cpu_x2, 1, &d_emlrtRTEI, true);
  emxInit_real32_T(&cpu_xp, 2, &emlrtRTEI, true);
  emxInit_creal32_T(&cpu_yp, 2, &emlrtRTEI, true);
  emxInit_real32_T(&cpu_fv, 2, &emlrtRTEI, true);
  emxInit_creal32_T(&b_cpu_yp, 2, &x_emlrtRTEI, true);
  emxInit_real32_T(&cpu_y, 2, &y_emlrtRTEI, true);
  emxInit_real32_T(&cpu_opts_filter, 1, &g_emlrtRTEI, true);
  emxInit_real32_T(&cpu_opts_x, 2, &g_emlrtRTEI, true);
  emxInit_real_T(&cpu_dv1, 1, &k_emlrtRTEI, true);
  emxInit_real_T(&cpu_dv2, 1, &k_emlrtRTEI, true);
  emxInit_real_T(&cpu_h1, 1, &l_emlrtRTEI, true);
  emxInit_real32_T(&cpu_h, 1, &m_emlrtRTEI, true);
  emxInit_uint32_T(&b_cpu_y, 2, &ab_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 2, &bb_emlrtRTEI, true);
  emxInit_real32_T(&cpu_y1, 2, &cb_emlrtRTEI, true);
  emxInit_real32_T(&yCol, 1, &db_emlrtRTEI, true);
  emxInit_real32_T(&cpu_w, 1, &v_emlrtRTEI, true);
  emxInit_real32_T(&c_cpu_y, 1, &v_emlrtRTEI, true);
  emxInit_real_T(&d_cpu_y, 2, &eb_emlrtRTEI, true);
  emxInit_real_T(&cpu_r, 2, &k_emlrtRTEI, true);
  emxInit_creal32_T(&c_cpu_yp, 2, &u_emlrtRTEI, true);
  for (int32_T p{0}; p < i1; p++) {
    int32_T b_c;
    int32_T vlen;
    int32_T x;
    uint32_T b_dv[2];
    boolean_T validLaunchParams;
    // 'ec_wtcc:28' [xx{p},yy{p}] =
    // wtc_lfn(x(:,c(p,1)),x(:,c(p,2)),fs,fLims,fVoices,ds);
    vlen = c->data[p];
    i = cpu_x1->size[0];
    cpu_x1->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real32_T(cpu_x1, i, &c_emlrtRTEI);
    b_c = c->data[p + c->size[0]];
    i = cpu_x2->size[0];
    cpu_x2->size[0] = cpu_x->size[0];
    emxEnsureCapacity_real32_T(cpu_x2, i, &d_emlrtRTEI);
    x = cpu_x->size[0] - 1;
    mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
    if (x_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    }
    x_needsGpuEnsureCapacity = false;
    gpuEmxEnsureCapacity_real32_T(cpu_x2, &gpu_x2, true);
    gpuEmxEnsureCapacity_real32_T(cpu_x1, &gpu_x1, true);
    if (x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
    }
    x_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_wtcc_kernel1<<<grid, block>>>(b_c, vlen, gpu_x, x, gpu_x2, gpu_x1,
                                       cpu_x->size[0U]);
    }
    //  Wavelet coherence - local function
    //  Calculate wavelet coherence
    // 'ec_wtcc:38' [xp,yp] =
    // wcoherence(x1,x2,fs,FrequencyLimits=fLims,VoicesPerOctave=fVoices);
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_x1, &gpu_x1);
    gpuEmxMemcpyGpuToCpu_real32_T(cpu_x2, &gpu_x2);
    coder::wcoherence(cpu_x1, cpu_x2, fs, fLims, fVoices, cpu_xp, cpu_yp,
                      &yp_outdatedOnCpu, &gpu_yp, &yp_outdatedOnGpu);
    // 'ec_wtcc:39' xp=xp';
    i = cpu_fv->size[0] * cpu_fv->size[1];
    cpu_fv->size[0] = cpu_xp->size[1];
    cpu_fv->size[1] = cpu_xp->size[0];
    emxEnsureCapacity_real32_T(cpu_fv, i, &e_emlrtRTEI);
    x = cpu_xp->size[0] - 1;
    i = cpu_xp->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(x, i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_xp, &gpu_xp, false);
    gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_xp, cpu_xp);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_wtcc_kernel2<<<grid, block>>>(gpu_xp, i, x, gpu_fv, cpu_fv->size[0U],
                                       cpu_xp->size[0U]);
    }
    fv_outdatedOnCpu = true;
    // 'ec_wtcc:39' yp=yp';
    i = b_cpu_yp->size[0] * b_cpu_yp->size[1];
    b_cpu_yp->size[0] = cpu_yp->size[1];
    b_cpu_yp->size[1] = cpu_yp->size[0];
    emxEnsureCapacity_creal32_T(b_cpu_yp, i, &f_emlrtRTEI);
    x = cpu_yp->size[0] - 1;
    i = cpu_yp->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(x, i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_yp, &gpu_yp, !yp_outdatedOnGpu);
    gpuEmxEnsureCapacity_creal32_T(b_cpu_yp, &b_gpu_yp, true);
    yp_needsGpuEnsureCapacity = false;
    if (yp_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_yp, cpu_yp);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_wtcc_kernel3<<<grid, block>>>(gpu_yp, i, x, b_gpu_yp,
                                       b_cpu_yp->size[0U], cpu_yp->size[0U]);
    }
    yp_outdatedOnGpu = false;
    b_yp_outdatedOnCpu = true;
    //  make column-major
    //  Downsample
    // 'ec_wtcc:42' if ds(2)
    if (ds[1] != 0.0) {
      real_T c_dv[4];
      real_T fc;
      real_T nZeroBegin;
      real_T opts_filterDelay;
      real_T opts_p;
      real_T pqmax;
      int32_T opts_dim;
      boolean_T opts_x_outdatedOnCpu;
      // 'ec_wtcc:43' xp = resample(xp,ds(1),ds(2));
      if ((cpu_fv->size[0] == 1) || (cpu_fv->size[1] == 1)) {
        opts_dim = 1;
      } else {
        opts_dim = 2;
        if (cpu_fv->size[0] != 1) {
          opts_dim = 1;
        }
      }
      opts_isRowVectorInput = (cpu_fv->size[0] == 1);
      if (opts_dim == 1) {
        if ((cpu_fv->size[0] == 1) || (cpu_fv->size[1] == 1)) {
          if ((cpu_fv->size[0] == 0) || (cpu_fv->size[1] == 0)) {
            b_c = 0;
          } else {
            vlen = cpu_fv->size[0];
            b_c = cpu_fv->size[1];
            if (vlen >= b_c) {
              b_c = vlen;
            }
          }
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
          cpu_opts_x->size[0] = b_c;
          cpu_opts_x->size[1] = 1;
          emxEnsureCapacity_real32_T(cpu_opts_x, i, &g_emlrtRTEI);
          mwGetLaunchParameters1D(computeNumIters(b_c - 1), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_opts_x, &gpu_opts_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel4<<<grid, block>>>(gpu_fv, b_c - 1, gpu_opts_x);
          }
          opts_x_outdatedOnCpu = true;
        } else {
          i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
          cpu_opts_x->size[0] = cpu_fv->size[0];
          cpu_opts_x->size[1] = cpu_fv->size[1];
          emxEnsureCapacity_real32_T(cpu_opts_x, i, &g_emlrtRTEI);
          i = cpu_fv->size[0] * cpu_fv->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_opts_x, &gpu_opts_x, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel24<<<grid, block>>>(gpu_fv, i, gpu_opts_x);
          }
          opts_x_outdatedOnCpu = true;
        }
      } else {
        i = cpu_opts_x->size[0] * cpu_opts_x->size[1];
        cpu_opts_x->size[0] = cpu_fv->size[1];
        cpu_opts_x->size[1] = cpu_fv->size[0];
        emxEnsureCapacity_real32_T(cpu_opts_x, i, &g_emlrtRTEI);
        i = cpu_fv->size[0] - 1;
        x = cpu_fv->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(i, x), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real32_T(cpu_opts_x, &gpu_opts_x, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_wtcc_kernel25<<<grid, block>>>(
              gpu_fv, x, i, gpu_opts_x, cpu_opts_x->size[0U], cpu_fv->size[0U]);
        }
        opts_x_outdatedOnCpu = true;
      }
      opts_p = coder::rat(ds[0] / ds[1], &opts_q);
      pqmax = std::fmax(opts_p, opts_q);
      fc = 0.5 / pqmax;
      pqmax = 20.0 * pqmax + 1.0;
      c_dv[0] = 0.0;
      c_dv[1] = 2.0 * fc;
      c_dv[2] = 2.0 * fc;
      c_dv[3] = 1.0;
      coder::firls(pqmax - 1.0, c_dv, cpu_r);
      i = cpu_dv1->size[0];
      cpu_dv1->size[0] = cpu_r->size[1];
      emxEnsureCapacity_real_T(cpu_dv1, i, &k_emlrtRTEI);
      x = cpu_r->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_r, &gpu_r, false);
      gpuEmxEnsureCapacity_real_T(cpu_dv1, &gpu_dv1, true);
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_r, cpu_r);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_wtcc_kernel5<<<grid, block>>>(gpu_r, x, gpu_dv1);
      }
      coder::kaiser(pqmax, cpu_dv2);
      i = cpu_h1->size[0];
      cpu_h1->size[0] = cpu_dv1->size[0];
      emxEnsureCapacity_real_T(cpu_h1, i, &l_emlrtRTEI);
      x = cpu_dv1->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_dv2, &gpu_dv2, false);
      gpuEmxEnsureCapacity_real_T(cpu_h1, &gpu_h1, true);
      yp_needsGpuEnsureCapacity = false;
      gpuEmxMemcpyCpuToGpu_real_T(&gpu_dv2, cpu_dv2);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_wtcc_kernel6<<<grid, block>>>(gpu_dv2, gpu_dv1, x, gpu_h1);
      }
      vlen = cpu_h1->size[0];
      if (cpu_h1->size[0] == 0) {
        pqmax = 0.0;
      } else {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_h1, &gpu_h1);
        pqmax = cpu_h1->data[0];
        yp_needsGpuEnsureCapacity = true;
        for (i = 0; i <= vlen - 2; i++) {
          pqmax += cpu_h1->data[i + 1];
        }
      }
      x = cpu_h1->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
      if (yp_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_real_T(cpu_h1, &gpu_h1, true);
      }
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_wtcc_kernel7<<<grid, block>>>(pqmax, opts_p, x, gpu_h1);
      }
      i = cpu_opts_filter->size[0];
      cpu_opts_filter->size[0] = cpu_h1->size[0];
      emxEnsureCapacity_real32_T(cpu_opts_filter, i, &g_emlrtRTEI);
      x = cpu_h1->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_opts_filter, &gpu_opts_filter, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_wtcc_kernel8<<<grid, block>>>(gpu_h1, x, gpu_opts_filter);
      }
      pqmax = (static_cast<real_T>(cpu_opts_filter->size[0]) - 1.0) / 2.0;
      nZeroBegin = std::floor(opts_q - coder::b_mod(pqmax, opts_q));
      opts_filterDelay = std::floor(std::ceil(pqmax + nZeroBegin) / opts_q);
      pqmax = static_cast<real_T>(cpu_opts_filter->size[0]) + nZeroBegin;
      fc = 0.0;
      vlen = cpu_opts_x->size[0];
      while (std::ceil(
                 (((static_cast<real_T>(vlen) - 1.0) * opts_p + pqmax) + fc) /
                 opts_q) -
                 opts_filterDelay <
             std::ceil(static_cast<real_T>(vlen) * opts_p / opts_q)) {
        fc++;
      }
      i = cpu_h->size[0];
      cpu_h->size[0] = static_cast<int32_T>(
          (nZeroBegin + static_cast<real_T>(cpu_opts_filter->size[0])) + fc);
      emxEnsureCapacity_real32_T(cpu_h, i, &m_emlrtRTEI);
      x = static_cast<int32_T>(
              (nZeroBegin + static_cast<real_T>(cpu_opts_filter->size[0])) +
              fc) -
          1;
      mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_h, &gpu_h, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_wtcc_kernel9<<<grid, block>>>(x, gpu_h);
      }
      if (cpu_opts_filter->size[0] < 1) {
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = 0;
      } else {
        uint32_T u;
        u = static_cast<uint32_T>(cpu_opts_filter->size[0]);
        i = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_opts_filter->size[0];
        emxEnsureCapacity_uint32_T(b_cpu_y, i, &n_emlrtRTEI);
        mwGetLaunchParameters1D(computeNumIters(static_cast<int32_T>(u) - 1),
                                &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_uint32_T(b_cpu_y, &gpu_y, true);
        y_needsGpuEnsureCapacity = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          ec_wtcc_kernel23<<<grid, block>>>(static_cast<int32_T>(u) - 1, gpu_y);
        }
      }
      i = cpu_iv1->size[0] * cpu_iv1->size[1];
      cpu_iv1->size[0] = 1;
      cpu_iv1->size[1] = b_cpu_y->size[1];
      emxEnsureCapacity_int32_T(cpu_iv1, i, &o_emlrtRTEI);
      x = b_cpu_y->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
      if (y_needsGpuEnsureCapacity) {
        gpuEmxEnsureCapacity_uint32_T(b_cpu_y, &gpu_y, true);
      }
      y_needsGpuEnsureCapacity = false;
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_wtcc_kernel10<<<grid, block>>>(gpu_y, nZeroBegin, x, gpu_iv1);
      }
      x = cpu_iv1->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(x), &grid, &block, 2147483647U);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        ec_wtcc_kernel11<<<grid, block>>>(gpu_opts_filter, gpu_iv1, x, gpu_h);
      }
      h_outdatedOnGpu = false;
      h_outdatedOnCpu = true;
      if (opts_dim == 1) {
        if (opts_isRowVectorInput) {
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = cpu_opts_x->size[0];
          cpu_y1->size[1] = cpu_opts_x->size[1];
          emxEnsureCapacity_real32_T(cpu_y1, i, &r_emlrtRTEI);
          x = cpu_opts_x->size[0] * cpu_opts_x->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(x), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_y1, &gpu_y1, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel12<<<grid, block>>>(gpu_opts_x, x, gpu_y1);
          }
          opts_isRowVectorInput = false;
          yp_needsGpuEnsureCapacity = true;
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_y1, &yp_needsGpuEnsureCapacity, &gpu_y1,
              &opts_isRowVectorInput, opts_p, opts_q, cpu_h, &h_outdatedOnCpu,
              &gpu_h, &h_outdatedOnGpu, opts_filterDelay);
          if ((cpu_y1->size[0] == 0) || (cpu_y1->size[1] == 0)) {
            b_c = 0;
          } else {
            vlen = cpu_y1->size[0];
            b_c = cpu_y1->size[1];
            if (vlen >= b_c) {
              b_c = vlen;
            }
          }
          i = cpu_fv->size[0] * cpu_fv->size[1];
          cpu_fv->size[0] = 1;
          cpu_fv->size[1] = b_c;
          emxEnsureCapacity_real32_T(cpu_fv, i, &q_emlrtRTEI);
          mwGetLaunchParameters1D(computeNumIters(b_c - 1), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_y1, &gpu_y1,
                                        !opts_isRowVectorInput);
          gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
          if (opts_isRowVectorInput) {
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_y1, cpu_y1);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel13<<<grid, block>>>(gpu_y1, b_c - 1, gpu_fv);
          }
        } else {
          i = cpu_fv->size[0] * cpu_fv->size[1];
          cpu_fv->size[0] = cpu_opts_x->size[0];
          cpu_fv->size[1] = cpu_opts_x->size[1];
          emxEnsureCapacity_real32_T(cpu_fv, i, &q_emlrtRTEI);
          x = cpu_opts_x->size[0] * cpu_opts_x->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(x), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel15<<<grid, block>>>(gpu_opts_x, x, gpu_fv);
          }
          yp_needsGpuEnsureCapacity = false;
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_fv, &fv_outdatedOnCpu, &gpu_fv, &yp_needsGpuEnsureCapacity,
              opts_p, opts_q, cpu_h, &h_outdatedOnCpu, &gpu_h, &h_outdatedOnGpu,
              opts_filterDelay);
        }
      } else {
        if (cpu_opts_x->size[0] == 1) {
          pqmax = std::ceil(opts_p / opts_q);
          opts_isRowVectorInput = false;
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = static_cast<int32_T>(pqmax);
          cpu_y1->size[1] = cpu_opts_x->size[1];
          emxEnsureCapacity_real32_T(cpu_y1, i, &s_emlrtRTEI);
          yp_needsGpuEnsureCapacity = true;
          opts_dim = cpu_opts_x->size[1];
          if (cpu_opts_x->size[1] - 1 >= 0) {
            if ((static_cast<int32_T>(opts_p) == 0) ||
                ((static_cast<int32_T>(opts_p) > 0) &&
                 (static_cast<int32_T>(opts_p) < 1))) {
              i3 = 1;
              i4 = 0;
            } else {
              i3 = static_cast<int32_T>(opts_p);
              i4 = static_cast<int32_T>(opts_p);
            }
            Ly = std::ceil(
                (0.0 * opts_p + static_cast<real_T>(cpu_h->size[0])) / opts_q);
          }
          for (x = 0; x < opts_dim; x++) {
            if (!(opts_p - 1.0 >= 0.0)) {
              y = nullptr;
              m = emlrtCreateCharArray(2, &iv2[0]);
              emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
              emlrtAssign(&y, m);
              b_y = nullptr;
              m1 = emlrtCreateDoubleScalar(opts_p - 1.0);
              emlrtAssign(&b_y, m1);
              emlrt_marshallIn(b_sprintf(y, b_y, &emlrtMCI),
                               "<output of sprintf>", unusedExpr);
            }
            i = yCol->size[0];
            yCol->size[0] = static_cast<int32_T>(opts_p);
            emxEnsureCapacity_real32_T(yCol, i, &t_emlrtRTEI);
            for (i = 0; i < static_cast<int32_T>(opts_p); i++) {
              yCol->data[i] = 0.0F;
            }
            vlen = div_s32(i4 - 1, i3) + 1;
            for (i = 0; i < vlen; i++) {
              if (opts_x_outdatedOnCpu) {
                gpuEmxMemcpyGpuToCpu_real32_T(cpu_opts_x, &gpu_opts_x);
              }
              opts_x_outdatedOnCpu = false;
              yCol->data[i3 * i] = cpu_opts_x->data[x];
            }
            b_yCol = *yCol;
            opts_x[0] = static_cast<int32_T>(opts_p);
            b_yCol.size = &opts_x[0];
            b_yCol.numDimensions = 1;
            coder::b_conv2(&b_yCol, cpu_h, &h_outdatedOnCpu, &gpu_h,
                           &h_outdatedOnGpu, cpu_w, &w_outdatedOnCpu, &gpu_w,
                           &w_outdatedOnGpu);
            if (!(opts_q - 1.0 >= 0.0)) {
              c_y = nullptr;
              m2 = emlrtCreateCharArray(2, &iv3[0]);
              emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
              emlrtAssign(&c_y, m2);
              d_y = nullptr;
              m3 = emlrtCreateDoubleScalar(opts_q - 1.0);
              emlrtAssign(&d_y, m3);
              emlrt_marshallIn(b_sprintf(c_y, d_y, &emlrtMCI),
                               "<output of sprintf>", unusedExpr);
            }
            if (opts_q == 0.0) {
              vlen = 1;
              b_c = -1;
            } else {
              vlen = static_cast<int32_T>(opts_q);
              b_c = cpu_w->size[0] - 1;
            }
            i = c_cpu_y->size[0];
            c_cpu_y->size[0] = div_s32(b_c, vlen) + 1;
            emxEnsureCapacity_real32_T(c_cpu_y, i, &v_emlrtRTEI);
            i = b_c / vlen;
            mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                    2147483647U);
            gpuEmxEnsureCapacity_real32_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
            gpuEmxEnsureCapacity_real32_T(c_cpu_y, &c_gpu_y, true);
            yp_outdatedOnGpu = false;
            if (w_outdatedOnGpu) {
              gpuEmxMemcpyCpuToGpu_real32_T(&gpu_w, cpu_w);
            }
            w_outdatedOnGpu = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              ec_wtcc_kernel16<<<grid, block>>>(gpu_w, vlen, i, c_gpu_y);
            }
            if (!(div_s32(b_c, vlen) + 1 < Ly)) {
              i = c_cpu_y->size[0];
              if (Ly < 1.0) {
                c_cpu_y->size[0] = 0;
              } else {
                c_cpu_y->size[0] = static_cast<int32_T>(Ly);
              }
              emxEnsureCapacity_real32_T(c_cpu_y, i, &w_emlrtRTEI);
              yp_outdatedOnGpu = true;
            }
            if (std::isnan(pqmax)) {
              i = d_cpu_y->size[0] * d_cpu_y->size[1];
              d_cpu_y->size[0] = 1;
              d_cpu_y->size[1] = 1;
              emxEnsureCapacity_real_T(d_cpu_y, i, &n_emlrtRTEI);
              gpuEmxEnsureCapacity_real_T(d_cpu_y, &d_gpu_y, true);
              b_y_needsGpuEnsureCapacity = false;
              ec_wtcc_kernel17<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
                  d_gpu_y);
            } else if (pqmax < 1.0) {
              d_cpu_y->size[0] = 1;
              d_cpu_y->size[1] = 0;
            } else {
              i = d_cpu_y->size[0] * d_cpu_y->size[1];
              d_cpu_y->size[0] = 1;
              d_cpu_y->size[1] = static_cast<int32_T>(pqmax - 1.0) + 1;
              emxEnsureCapacity_real_T(d_cpu_y, i, &n_emlrtRTEI);
              mwGetLaunchParameters1D(
                  computeNumIters(static_cast<int32_T>(pqmax - 1.0)), &grid,
                  &block, 2147483647U);
              gpuEmxEnsureCapacity_real_T(d_cpu_y, &d_gpu_y, true);
              b_y_needsGpuEnsureCapacity = false;
              validLaunchParams = mwValidateLaunchParameters(grid, block);
              if (validLaunchParams) {
                ec_wtcc_kernel19<<<grid, block>>>(
                    static_cast<int32_T>(pqmax - 1.0), d_gpu_y);
              }
            }
            vlen = cpu_y1->size[0];
            mwGetLaunchParameters1D(computeNumIters(vlen - 1), &grid, &block,
                                    2147483647U);
            if (yp_outdatedOnGpu) {
              gpuEmxEnsureCapacity_real32_T(c_cpu_y, &c_gpu_y, true);
            }
            if (b_y_needsGpuEnsureCapacity) {
              gpuEmxEnsureCapacity_real_T(d_cpu_y, &d_gpu_y, true);
            }
            b_y_needsGpuEnsureCapacity = false;
            if (yp_needsGpuEnsureCapacity) {
              gpuEmxEnsureCapacity_real32_T(cpu_y1, &gpu_y1, true);
            }
            yp_needsGpuEnsureCapacity = false;
            validLaunchParams = mwValidateLaunchParameters(grid, block);
            if (validLaunchParams) {
              ec_wtcc_kernel18<<<grid, block>>>(c_gpu_y, d_gpu_y,
                                                opts_filterDelay, x, vlen - 1,
                                                gpu_y1, cpu_y1->size[0U]);
            }
          }
        } else {
          i = cpu_y1->size[0] * cpu_y1->size[1];
          cpu_y1->size[0] = cpu_opts_x->size[0];
          cpu_y1->size[1] = cpu_opts_x->size[1];
          emxEnsureCapacity_real32_T(cpu_y1, i, &p_emlrtRTEI);
          x = cpu_opts_x->size[0] * cpu_opts_x->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(x), &grid, &block,
                                  2147483647U);
          gpuEmxEnsureCapacity_real32_T(cpu_y1, &gpu_y1, true);
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel22<<<grid, block>>>(gpu_opts_x, x, gpu_y1);
          }
          opts_isRowVectorInput = false;
          yp_needsGpuEnsureCapacity = true;
          coder::b_signal::internal::resample::uniformResampleAlongFirstDim(
              cpu_y1, &yp_needsGpuEnsureCapacity, &gpu_y1,
              &opts_isRowVectorInput, opts_p, opts_q, cpu_h, &h_outdatedOnCpu,
              &gpu_h, &h_outdatedOnGpu, opts_filterDelay);
          yp_needsGpuEnsureCapacity = true;
        }
        if ((cpu_opts_x->size[0] == 1) || (cpu_opts_x->size[1] == 1)) {
          vlen = cpu_fv->size[0];
          b_c = cpu_fv->size[0];
          opts_dim = cpu_y1->size[0];
          i = cpu_fv->size[0] * cpu_fv->size[1];
          cpu_fv->size[0] = vlen;
          cpu_fv->size[1] = cpu_y1->size[0];
          emxEnsureCapacity_real32_T(cpu_fv, i, &q_emlrtRTEI);
          i = b_c * opts_dim - 1;
          mwGetLaunchParameters1D(computeNumIters(i), &grid, &block,
                                  2147483647U);
          if (yp_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real32_T(cpu_y1, &gpu_y1,
                                          !opts_isRowVectorInput);
          }
          gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
          if (opts_isRowVectorInput) {
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_y1, cpu_y1);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel20<<<grid, block>>>(gpu_y1, i, gpu_fv);
          }
        } else {
          i = cpu_fv->size[0] * cpu_fv->size[1];
          cpu_fv->size[0] = cpu_y1->size[1];
          cpu_fv->size[1] = cpu_y1->size[0];
          emxEnsureCapacity_real32_T(cpu_fv, i, &q_emlrtRTEI);
          x = cpu_y1->size[0] - 1;
          i = cpu_y1->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(x, i), &grid, &block,
                                  2147483647U);
          if (yp_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_real32_T(cpu_y1, &gpu_y1,
                                          !opts_isRowVectorInput);
          }
          gpuEmxEnsureCapacity_real32_T(cpu_fv, &gpu_fv, true);
          if (opts_isRowVectorInput) {
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_y1, cpu_y1);
          }
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            ec_wtcc_kernel21<<<grid, block>>>(
                gpu_y1, i, x, gpu_fv, cpu_fv->size[0U], cpu_y1->size[0U]);
          }
        }
      }
      // 'ec_wtcc:44' yp = resample(yp,ds(1),ds(2));
      opts_isRowVectorInput = false;
      i = c_cpu_yp->size[0] * c_cpu_yp->size[1];
      c_cpu_yp->size[0] = b_cpu_yp->size[0];
      c_cpu_yp->size[1] = b_cpu_yp->size[1];
      emxEnsureCapacity_creal32_T(c_cpu_yp, i, &u_emlrtRTEI);
      vlen = b_cpu_yp->size[0] * b_cpu_yp->size[1] - 1;
      for (i = 0; i <= vlen; i++) {
        if (b_yp_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_creal32_T(b_cpu_yp, &b_gpu_yp);
        }
        b_yp_outdatedOnCpu = false;
        c_cpu_yp->data[i] = b_cpu_yp->data[i];
        opts_isRowVectorInput = true;
      }
      coder::resample(c_cpu_yp, &c_gpu_yp, &opts_isRowVectorInput, ds[0], ds[1],
                      b_cpu_yp, &b_yp_outdatedOnCpu, &b_gpu_yp,
                      &yp_outdatedOnGpu);
      yp_needsGpuEnsureCapacity = true;
    }
    //  Convert wavelet to phase angle
    // 'ec_wtcc:48' yp = angle(yp);
    vlen = b_cpu_yp->size[0] * b_cpu_yp->size[1];
    for (i = 0; i < 2; i++) {
      b_dv[i] = static_cast<uint32_T>(b_cpu_yp->size[i]);
    }
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = static_cast<int32_T>(b_dv[0]);
    cpu_y->size[1] = static_cast<int32_T>(b_dv[1]);
    emxEnsureCapacity_real32_T(cpu_y, i, &h_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(vlen - 1), &grid, &block,
                            2147483647U);
    if (yp_needsGpuEnsureCapacity) {
      gpuEmxEnsureCapacity_creal32_T(b_cpu_yp, &b_gpu_yp, !yp_outdatedOnGpu);
    }
    gpuEmxEnsureCapacity_real32_T(cpu_y, &b_gpu_y, true);
    if (yp_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(&b_gpu_yp, b_cpu_yp);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      ec_wtcc_kernel14<<<grid, block>>>(b_gpu_yp, vlen - 1, b_gpu_y);
    }
    yp_needsGpuEnsureCapacity = true;
    i = yy->data[p].f1->size[0] * yy->data[p].f1->size[1];
    yy->data[p].f1->size[0] = cpu_y->size[0];
    emxEnsureCapacity_real32_T(yy->data[p].f1, i, &i_emlrtRTEI);
    i = yy->data[p].f1->size[0] * yy->data[p].f1->size[1];
    yy->data[p].f1->size[1] = cpu_y->size[1];
    emxEnsureCapacity_real32_T(yy->data[p].f1, i, &i_emlrtRTEI);
    for (i = 0; i < cpu_y->size[0] * cpu_y->size[1]; i++) {
      if (yp_needsGpuEnsureCapacity) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &b_gpu_y);
      }
      yp_needsGpuEnsureCapacity = false;
      yy->data[p].f1->data[i] = cpu_y->data[i];
    }
    i = xx->data[p].f1->size[0] * xx->data[p].f1->size[1];
    xx->data[p].f1->size[0] = cpu_fv->size[0];
    emxEnsureCapacity_real32_T(xx->data[p].f1, i, &j_emlrtRTEI);
    i = xx->data[p].f1->size[0] * xx->data[p].f1->size[1];
    xx->data[p].f1->size[1] = cpu_fv->size[1];
    emxEnsureCapacity_real32_T(xx->data[p].f1, i, &j_emlrtRTEI);
    for (i = 0; i < cpu_fv->size[0] * cpu_fv->size[1]; i++) {
      if (fv_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_fv, &gpu_fv);
      }
      fv_outdatedOnCpu = false;
      xx->data[p].f1->data[i] = cpu_fv->data[i];
    }
  }
  emxFree_creal32_T(&c_cpu_yp);
  emxFree_real_T(&cpu_r);
  emxFree_real_T(&d_cpu_y);
  emxFree_real32_T(&c_cpu_y);
  emxFree_real32_T(&cpu_w);
  emxFree_real32_T(&yCol);
  emxFree_real32_T(&cpu_y1);
  emxFree_int32_T(&cpu_iv1);
  emxFree_uint32_T(&b_cpu_y);
  emxFree_real32_T(&cpu_h);
  emxFree_real_T(&cpu_h1);
  emxFree_real_T(&cpu_dv2);
  emxFree_real_T(&cpu_dv1);
  emxFree_real32_T(&cpu_opts_x);
  emxFree_real32_T(&cpu_opts_filter);
  emxFree_real32_T(&cpu_y);
  emxFree_creal32_T(&b_cpu_yp);
  emxFree_real32_T(&cpu_fv);
  emxFree_creal32_T(&cpu_yp);
  emxFree_real32_T(&cpu_xp);
  emxFree_real32_T(&cpu_x2);
  emxFree_real32_T(&cpu_x1);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_x);
  gpuEmxFree_real32_T(&gpu_x1);
  gpuEmxFree_real32_T(&gpu_x2);
  gpuEmxFree_real32_T(&gpu_xp);
  gpuEmxFree_creal32_T(&gpu_yp);
  gpuEmxFree_real32_T(&gpu_fv);
  gpuEmxFree_creal32_T(&b_gpu_yp);
  gpuEmxFree_real32_T(&b_gpu_y);
  gpuEmxFree_real32_T(&gpu_opts_filter);
  gpuEmxFree_real32_T(&gpu_opts_x);
  gpuEmxFree_real_T(&gpu_dv1);
  gpuEmxFree_real_T(&gpu_dv2);
  gpuEmxFree_real_T(&gpu_h1);
  gpuEmxFree_real32_T(&gpu_h);
  gpuEmxFree_uint32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_real32_T(&gpu_y1);
  gpuEmxFree_real32_T(&gpu_w);
  gpuEmxFree_real32_T(&c_gpu_y);
  gpuEmxFree_real_T(&d_gpu_y);
  gpuEmxFree_real_T(&gpu_r);
  gpuEmxFree_creal32_T(&c_gpu_yp);
}

// End of code generation (ec_wtcc.cu)

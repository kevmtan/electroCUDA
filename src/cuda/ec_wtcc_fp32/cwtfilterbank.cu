#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtfilterbank.cu
//
// Code generation for function 'cwtfilterbank'
//

// Include files
#include "cwtfilterbank.h"
#include "cwtfreqlimits.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "log2.h"
#include "rt_nonfinite.h"
#include "wavCFandSD.h"
#include "wavbpfilters.h"
#include <cmath>

// Variable Definitions
static emlrtMCInfo b_emlrtMCI{
    53,                                                                // lineNo
    14,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtMCInfo c_emlrtMCI{
    55,                                                                // lineNo
    15,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtRTEInfo wb_emlrtRTEI{
    999,             // lineNo
    17,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo xb_emlrtRTEI{
    1000,            // lineNo
    17,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo yb_emlrtRTEI{
    1009,            // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo ac_emlrtRTEI{
    1278,            // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo bc_emlrtRTEI{
    952,             // lineNo
    34,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo cc_emlrtRTEI{
    952,             // lineNo
    45,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo dc_emlrtRTEI{
    217,             // lineNo
    21,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo ec_emlrtRTEI{
    18,             // lineNo
    5,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo gc_emlrtRTEI{
    31,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo hc_emlrtRTEI{
    26,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo ic_emlrtRTEI{
    33,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo jc_emlrtRTEI{
    27,             // lineNo
    9,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo kc_emlrtRTEI{
    956,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo lc_emlrtRTEI{
    227,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo nc_emlrtRTEI{
    228,             // lineNo
    13,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo oc_emlrtRTEI{
    1276,            // lineNo
    30,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo pc_emlrtRTEI{
    1276,            // lineNo
    25,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo qc_emlrtRTEI{
    16,             // lineNo
    5,              // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo rc_emlrtRTEI{
    32,             // lineNo
    18,             // colNo
    "getCWTScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "getCWTScales.m" // pName
};

static emlrtRTEInfo sc_emlrtRTEI{
    32,             // lineNo
    13,             // colNo
    "getCWTScales", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "getCWTScales.m" // pName
};

static emlrtRTEInfo tc_emlrtRTEI{
    34,             // lineNo
    42,             // colNo
    "wavbpfilters", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/+wavelet/+internal/+cwt/"
    "wavbpfilters.m" // pName
};

static emlrtRTEInfo uc_emlrtRTEI{
    934,             // lineNo
    26,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

static emlrtRTEInfo vc_emlrtRTEI{
    816,             // lineNo
    25,              // colNo
    "cwtfilterbank", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/"
    "cwtfilterbank.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret);

namespace coder {
static cwtfilterbank *cwtfilterbank_setProperties(cwtfilterbank *self,
                                                  real_T varargin_2,
                                                  const real_T varargin_6[2],
                                                  real_T varargin_8,
                                                  real_T varargin_10);

}
static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y);

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location);

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret)
{
  static const int32_T dims[2]{1, -1};
  int32_T iv[2];
  int32_T i;
  boolean_T bv[2]{false, true};
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                            (const void *)&dims[0], &bv[0], &iv[0]);
  i = ret->size[0] * ret->size[1];
  ret->size[0] = iv[0];
  ret->size[1] = iv[1];
  emxEnsureCapacity_char_T(ret, i, static_cast<emlrtRTEInfo *>(nullptr));
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret->data[0], 1, false);
  emlrtDestroyArray(&src);
}

//
//
namespace coder {
static cwtfilterbank *cwtfilterbank_setProperties(cwtfilterbank *self,
                                                  real_T varargin_2,
                                                  const real_T varargin_6[2],
                                                  real_T varargin_8,
                                                  real_T varargin_10)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 2};
  static const int32_T iv2[2]{1, 7};
  static const int32_T iv3[2]{1, 5};
  static const char_T b[10]{'r', 'e', 'f', 'l', 'e', 'c', 't', 'i', 'o', 'n'};
  static const char_T cv2[10]{'r', 'e', 'f', 'l', 'e', 'c', 't', 'i', 'o', 'n'};
  static const char_T u[7]{'s', 'p', 'r', 'i', 'n', 't', 'f'};
  static const char_T b_formatSpec[5]{'%', '2', '.', '2', 'f'};
  static const char_T cv3[4]{'b', 'u', 'm', 'p'};
  static const char_T cv4[4]{'a', 'm', 'o', 'r'};
  static const char_T formatSpec[2]{'%', 'f'};
  cwtfilterbank *b_self;
  emxArray_char_T *d_tmpStr;
  emxArray_char_T *tmpStr;
  const mxArray *b_tmpStr;
  const mxArray *b_y;
  const mxArray *c_tmpStr;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *f_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *m4;
  const mxArray *m5;
  const mxArray *y;
  real_T NyquistRange_idx_1;
  real_T cf;
  real_T sigmat;
  int32_T exitg2;
  int32_T k;
  char_T a[10];
  boolean_T b_b[2];
  boolean_T exitg1;
  boolean_T freqsep;
  boolean_T guard1;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  b_self = self;
  b_self->CutOff = 50.0;
  b_self->Gamma = 3.0;
  b_self->Beta = 20.0;
  b_self->Wavelet[0] = 'a';
  b_self->Wavelet[1] = 'm';
  b_self->Wavelet[2] = 'o';
  b_self->Wavelet[3] = 'r';
  b_self->TimeBandwidth = rtNaN;
  b_self->SignalLength = varargin_2;
  NyquistRange_idx_1 = b_self->SignalLength / 2.0;
  NyquistRange_idx_1 = std::floor(NyquistRange_idx_1);
  b_self->SignalPad = NyquistRange_idx_1;
  b_self->VoicesPerOctave = varargin_10;
  b_self->SamplingFrequency = varargin_8;
  b_self->WaveletParameters[0] = rtNaN;
  b_self->FrequencyLimits[0] = varargin_6[0];
  b_self->WaveletParameters[1] = rtNaN;
  b_self->FrequencyLimits[1] = varargin_6[1];
  for (k = 0; k < 10; k++) {
    b_self->Boundary[k] = b[k];
  }
  NyquistRange_idx_1 = b_self->TimeBandwidth;
  guard1 = false;
  if (!std::isnan(NyquistRange_idx_1)) {
    NyquistRange_idx_1 = b_self->WaveletParameters[0];
    b_b[0] = std::isnan(NyquistRange_idx_1);
    NyquistRange_idx_1 = b_self->WaveletParameters[1];
    b_b[1] = std::isnan(NyquistRange_idx_1);
    freqsep = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k < 2)) {
      if (!b_b[k]) {
        freqsep = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    if (freqsep) {
      b_self->Beta = b_self->TimeBandwidth / b_self->Gamma;
    } else {
      guard1 = true;
    }
  } else {
    guard1 = true;
  }
  if (guard1) {
    NyquistRange_idx_1 = b_self->WaveletParameters[0];
    b_b[0] = std::isnan(NyquistRange_idx_1);
    NyquistRange_idx_1 = b_self->WaveletParameters[1];
    b_b[1] = std::isnan(NyquistRange_idx_1);
    freqsep = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k < 2)) {
      if (!b_b[k]) {
        freqsep = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    if (!freqsep) {
      NyquistRange_idx_1 = b_self->TimeBandwidth;
      if (std::isnan(NyquistRange_idx_1)) {
        b_self->Gamma = b_self->WaveletParameters[0];
        b_self->Beta = b_self->WaveletParameters[1] / b_self->Gamma;
      }
    }
  }
  for (k = 0; k < 10; k++) {
    a[k] = b_self->Boundary[k];
  }
  freqsep = false;
  k = 0;
  do {
    exitg2 = 0;
    if (k + 1 < 11) {
      if (cv[static_cast<uint8_T>(a[k]) & 127] !=
          cv[static_cast<int32_T>(cv2[k])]) {
        exitg2 = 1;
      } else {
        k++;
      }
    } else {
      freqsep = true;
      exitg2 = 1;
    }
  } while (exitg2 == 0);
  if (freqsep) {
    if (b_self->SignalLength <= 100000.0) {
      NyquistRange_idx_1 = b_self->SignalLength / 2.0;
      NyquistRange_idx_1 = std::floor(NyquistRange_idx_1);
      b_self->SignalPad = NyquistRange_idx_1;
    } else {
      NyquistRange_idx_1 = b_log2(b_self->SignalLength);
      NyquistRange_idx_1 = std::ceil(NyquistRange_idx_1);
      b_self->SignalPad = NyquistRange_idx_1;
    }
  } else {
    b_self->SignalPad = 0.0;
  }
  NyquistRange_idx_1 = b_self->FrequencyLimits[0];
  b_b[0] = std::isnan(NyquistRange_idx_1);
  NyquistRange_idx_1 = b_self->FrequencyLimits[1];
  b_b[1] = std::isnan(NyquistRange_idx_1);
  freqsep = true;
  k = 0;
  exitg1 = false;
  while ((!exitg1) && (k < 2)) {
    if (!b_b[k]) {
      freqsep = false;
      exitg1 = true;
    } else {
      k++;
    }
  }
  if (!freqsep) {
    real_T FourierFactor;
    real_T be;
    real_T cutoff;
    real_T freqrange_idx_0;
    real_T freqrange_idx_1;
    real_T fs;
    real_T ga;
    real_T omegac;
    real_T varargin_1;
    real_T varargin_3;
    char_T b_wav[4];
    char_T wav[4];
    char_T c;
    freqrange_idx_0 = b_self->FrequencyLimits[0];
    freqrange_idx_1 = b_self->FrequencyLimits[1];
    NyquistRange_idx_1 = b_self->SamplingFrequency / 2.0;
    emxInit_char_T(&tmpStr, 2, &vc_emlrtRTEI, true);
    if ((freqrange_idx_1 <= 0.0) || (freqrange_idx_0 >= NyquistRange_idx_1)) {
      b_tmpStr = nullptr;
      y = nullptr;
      m = emlrtCreateCharArray(2, &iv[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &u[0]);
      emlrtAssign(&y, m);
      b_y = nullptr;
      m1 = emlrtCreateCharArray(2, &iv1[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 2, m1, &formatSpec[0]);
      emlrtAssign(&b_y, m1);
      c_y = nullptr;
      m2 = emlrtCreateDoubleScalar(NyquistRange_idx_1);
      emlrtAssign(&c_y, m2);
      emlrtAssign(&b_tmpStr, feval(y, b_y, c_y, &b_emlrtMCI));
      emlrt_marshallIn(length(emlrtAlias(b_tmpStr), &c_emlrtMCI),
                       "<output of length>");
      emlrt_marshallIn(emlrtAlias(b_tmpStr), "tmpStr", tmpStr);
      emlrtDestroyArray(&b_tmpStr);
    }
    emxFree_char_T(&tmpStr);
    fs = b_self->SamplingFrequency;
    ga = b_self->Gamma;
    be = b_self->Beta;
    NyquistRange_idx_1 = b_self->SignalLength;
    varargin_3 = b_self->VoicesPerOctave;
    cutoff = b_self->CutOff;
    varargin_1 = b_self->SamplingFrequency;
    c = b_self->Wavelet[0];
    wav[0] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[1];
    wav[1] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[2];
    wav[2] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[3];
    wav[3] = cv[static_cast<uint8_T>(c) & 127];
    omegac = 3.1415926535897931;
    cutoff /= 100.0;
    b_wav[0] = wav[0];
    b_wav[1] = wav[1];
    b_wav[2] = wav[2];
    b_wav[3] = wav[3];
    FourierFactor =
        wavelet::internal::cwt::wavCFandSD(b_wav, ga, be, &sigmat, &cf);
    sigmat = NyquistRange_idx_1 / (sigmat * 2.0);
    k = 0;
    do {
      exitg2 = 0;
      if (k + 1 < 5) {
        if (cv3[k] != wav[k]) {
          exitg2 = 1;
        } else {
          k++;
        }
      } else {
        freqsep = true;
        exitg2 = 1;
      }
    } while (exitg2 == 0);
    if (freqsep) {
      k = 1;
    } else {
      k = 0;
      do {
        exitg2 = 0;
        if (k + 1 < 5) {
          if (cv4[k] != wav[k]) {
            exitg2 = 1;
          } else {
            k++;
          }
        } else {
          freqsep = true;
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (freqsep) {
        k = 2;
      } else {
        k = -1;
      }
    }
    switch (k) {
    case 0:
      omegac =
          wavelet::internal::cwt::getFreqFromCutoffMorse(cutoff, cf, ga, be);
      break;
    case 1:
      omegac = wavelet::internal::cwt::getFreqFromCutoffBump(cutoff, cf);
      break;
    case 2:
      omegac = wavelet::internal::cwt::getFreqFromCutoffAmor(cutoff, cf);
      break;
    }
    NyquistRange_idx_1 = omegac / 3.1415926535897931;
    if (sigmat < NyquistRange_idx_1 * rt_powd_snf(2.0, 1.0 / varargin_3)) {
      sigmat = NyquistRange_idx_1 * rt_powd_snf(2.0, 1.0 / varargin_3);
    }
    NyquistRange_idx_1 = 1.0 / (sigmat * FourierFactor) * varargin_1;
    if (freqrange_idx_0 < NyquistRange_idx_1) {
      b_self->FrequencyLimits[0] = NyquistRange_idx_1;
      freqrange_idx_0 = b_self->FrequencyLimits[0];
    }
    if (freqrange_idx_1 > fs / 2.0) {
      b_self->FrequencyLimits[1] = fs / 2.0;
      freqrange_idx_1 = b_self->FrequencyLimits[1];
    }
    freqsep = (b_log2(freqrange_idx_1) - b_log2(freqrange_idx_0) >=
               1.0 / b_self->VoicesPerOctave);
    if (!freqsep) {
      varargin_1 = 1.0 / b_self->VoicesPerOctave;
      c_tmpStr = nullptr;
      d_y = nullptr;
      m3 = emlrtCreateCharArray(2, &iv2[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m3, &u[0]);
      emlrtAssign(&d_y, m3);
      e_y = nullptr;
      m4 = emlrtCreateCharArray(2, &iv3[0]);
      emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m4, &b_formatSpec[0]);
      emlrtAssign(&e_y, m4);
      f_y = nullptr;
      m5 = emlrtCreateDoubleScalar(varargin_1);
      emlrtAssign(&f_y, m5);
      emlrtAssign(&c_tmpStr, feval(d_y, e_y, f_y, &b_emlrtMCI));
      emlrt_marshallIn(length(emlrtAlias(c_tmpStr), &c_emlrtMCI),
                       "<output of length>");
      emxInit_char_T(&d_tmpStr, 2, &vc_emlrtRTEI, true);
      emlrt_marshallIn(emlrtAlias(c_tmpStr), "tmpStr", d_tmpStr);
      emxFree_char_T(&d_tmpStr);
      emlrtDestroyArray(&c_tmpStr);
    }
  }
  b_self->CutOff = 10.0;
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  return b_self;
}

} // namespace coder
static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(tmpStr), &thisId, y);
  emlrtDestroyArray(&tmpStr);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y)
{
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location)
{
  const mxArray *pArrays[3];
  const mxArray *m;
  pArrays[0] = m1;
  pArrays[1] = m2;
  pArrays[2] = m3;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 3, &pArrays[0],
                               "feval", true, location);
}

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location)
{
  const mxArray *m;
  const mxArray *pArray;
  pArray = m1;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 1, &pArray, "length",
                               true, location);
}

//
//
namespace coder {
cwtfilterbank *cwtfilterbank_cwtfilterbank(cwtfilterbank *self,
                                           real_T varargin_2,
                                           const real_T varargin_6[2],
                                           real_T varargin_8,
                                           real_T varargin_10)
{
  static const char_T cv1[4]{'b', 'u', 'm', 'p'};
  static const char_T cv2[4]{'a', 'm', 'o', 'r'};
  cwtfilterbank *b_self;
  emxArray_real_T *b_y;
  emxArray_real_T *c_b;
  emxArray_real_T *c_y;
  emxArray_real_T *d_b;
  emxArray_real_T *d_y;
  emxArray_real_T *daughter;
  emxArray_real_T *expnt;
  emxArray_real_T *f;
  emxArray_real_T *omega;
  emxArray_real_T *omega_tmp1;
  emxArray_real_T *omega_tmp2;
  emxArray_real_T *scales;
  emxArray_real_T *somega;
  emxArray_real_T *w;
  real_T N;
  real_T b;
  real_T cf;
  real_T ga;
  real_T nbSamp;
  int32_T b_i;
  int32_T exitg2;
  int32_T fc;
  int32_T i;
  int32_T loop_ub;
  int32_T nx;
  char_T wname[4];
  boolean_T b_b[2];
  boolean_T exitg1;
  boolean_T y;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  b_self = self;
  b_self = cwtfilterbank_setProperties(b_self, varargin_2, varargin_6,
                                       varargin_8, varargin_10);
  N = b_self->SignalLength + 2.0 * b_self->SignalPad;
  b = std::trunc(N / 2.0);
  emxInit_real_T(&omega_tmp1, 2, &wb_emlrtRTEI, true);
  if (std::isnan(b)) {
    i = omega_tmp1->size[0] * omega_tmp1->size[1];
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = 1;
    emxEnsureCapacity_real_T(omega_tmp1, i, &wb_emlrtRTEI);
    omega_tmp1->data[0] = rtNaN;
  } else if (b < 1.0) {
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = 0;
  } else {
    i = omega_tmp1->size[0] * omega_tmp1->size[1];
    omega_tmp1->size[0] = 1;
    omega_tmp1->size[1] = static_cast<int32_T>(b - 1.0) + 1;
    emxEnsureCapacity_real_T(omega_tmp1, i, &wb_emlrtRTEI);
    for (i = 0; i <= static_cast<int32_T>(b - 1.0); i++) {
      omega_tmp1->data[i] = static_cast<real_T>(i) + 1.0;
    }
  }
  emxInit_real_T(&omega_tmp2, 2, &xb_emlrtRTEI, true);
  i = omega_tmp2->size[0] * omega_tmp2->size[1];
  omega_tmp2->size[0] = 1;
  omega_tmp2->size[1] = omega_tmp1->size[1];
  emxEnsureCapacity_real_T(omega_tmp2, i, &xb_emlrtRTEI);
  nbSamp = 6.2831853071795862 / N;
  for (i = 0; i < omega_tmp1->size[1]; i++) {
    omega_tmp2->data[i] = omega_tmp1->data[i] * nbSamp;
  }
  emxFree_real_T(&omega_tmp1);
  N = std::trunc((N - 1.0) / 2.0);
  if (N < 1.0) {
    b_i = 0;
    nx = 1;
    fc = -1;
  } else {
    b_i = static_cast<int32_T>(N) - 1;
    nx = -1;
    fc = 0;
  }
  i = b_self->Omega->size[0] * b_self->Omega->size[1];
  b_self->Omega->size[0] = 1;
  b_self->Omega->size[1] = (omega_tmp2->size[1] + div_s32(fc - b_i, nx)) + 2;
  emxEnsureCapacity_real_T(b_self->Omega, i, &yb_emlrtRTEI);
  b_self->Omega->data[0] = 0.0;
  loop_ub = omega_tmp2->size[1];
  for (i = 0; i < loop_ub; i++) {
    b_self->Omega->data[i + 1] = omega_tmp2->data[i];
  }
  loop_ub = div_s32(fc - b_i, nx);
  for (i = 0; i <= loop_ub; i++) {
    b_self->Omega->data[(i + omega_tmp2->size[1]) + 1] =
        -omega_tmp2->data[b_i + nx * i];
  }
  emxFree_real_T(&omega_tmp2);
  nbSamp = b_self->FrequencyLimits[0];
  b_b[0] = std::isnan(nbSamp);
  nbSamp = b_self->FrequencyLimits[1];
  b_b[1] = std::isnan(nbSamp);
  y = true;
  loop_ub = 0;
  exitg1 = false;
  while ((!exitg1) && (loop_ub < 2)) {
    if (!b_b[loop_ub]) {
      y = false;
      exitg1 = true;
    } else {
      loop_ub++;
    }
  }
  if (!y) {
    real_T frange_idx_0;
    real_T nv;
    char_T b_wname[4];
    nbSamp = b_self->FrequencyLimits[0];
    nbSamp /= b_self->SamplingFrequency;
    nbSamp = nbSamp * 2.0 * 3.1415926535897931;
    frange_idx_0 = nbSamp;
    nbSamp = b_self->FrequencyLimits[1];
    nbSamp /= b_self->SamplingFrequency;
    nbSamp = nbSamp * 2.0 * 3.1415926535897931;
    nv = b_self->VoicesPerOctave;
    ga = rt_powd_snf(2.0, 1.0 / nv);
    b_wname[0] = b_self->Wavelet[0];
    b_wname[1] = b_self->Wavelet[1];
    b_wname[2] = b_self->Wavelet[2];
    b_wname[3] = b_self->Wavelet[3];
    wavelet::internal::cwt::wavCFandSD(b_wname, b_self->Gamma, b_self->Beta, &b,
                                       &cf);
    N = cf / nbSamp;
    emxInit_real_T(&b_y, 2, &oc_emlrtRTEI, true);
    b = nv * b_log2(cf / frange_idx_0 / N);
    if (std::isnan(b)) {
      i = b_y->size[0] * b_y->size[1];
      b_y->size[0] = 1;
      b_y->size[1] = 1;
      emxEnsureCapacity_real_T(b_y, i, &n_emlrtRTEI);
      b_y->data[0] = rtNaN;
    } else if (b < 0.0) {
      b_y->size[0] = 1;
      b_y->size[1] = 0;
    } else {
      i = b_y->size[0] * b_y->size[1];
      b_y->size[0] = 1;
      b_y->size[1] = static_cast<int32_T>(b) + 1;
      emxEnsureCapacity_real_T(b_y, i, &n_emlrtRTEI);
      for (i = 0; i <= static_cast<int32_T>(b); i++) {
        b_y->data[i] = i;
      }
    }
    emxInit_real_T(&c_b, 2, &pc_emlrtRTEI, true);
    i = c_b->size[0] * c_b->size[1];
    c_b->size[0] = 1;
    c_b->size[1] = b_y->size[1];
    emxEnsureCapacity_real_T(c_b, i, &kb_emlrtRTEI);
    nx = b_y->size[1];
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      c_b->data[loop_ub] = rt_powd_snf(ga, b_y->data[loop_ub]);
    }
    emxFree_real_T(&b_y);
    i = b_self->Scales->size[0] * b_self->Scales->size[1];
    b_self->Scales->size[0] = 1;
    b_self->Scales->size[1] = c_b->size[1];
    emxEnsureCapacity_real_T(b_self->Scales, i, &ac_emlrtRTEI);
    for (i = 0; i < c_b->size[1]; i++) {
      b_self->Scales->data[i] = N * c_b->data[i];
    }
    emxFree_real_T(&c_b);
  } else {
    real_T be;
    real_T frange_idx_0;
    real_T nv;
    real_T omegac;
    char_T b_wname[4];
    char_T c;
    nbSamp = b_self->SignalLength;
    ga = b_self->Gamma;
    be = b_self->Beta;
    nv = b_self->VoicesPerOctave;
    N = b_self->CutOff;
    c = b_self->Wavelet[0];
    wname[0] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[1];
    wname[1] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[2];
    wname[2] = cv[static_cast<uint8_T>(c) & 127];
    c = b_self->Wavelet[3];
    wname[3] = cv[static_cast<uint8_T>(c) & 127];
    omegac = 3.1415926535897931;
    frange_idx_0 = N / 100.0;
    b_wname[0] = wname[0];
    b_wname[1] = wname[1];
    b_wname[2] = wname[2];
    b_wname[3] = wname[3];
    wavelet::internal::cwt::wavCFandSD(b_wname, ga, be, &b, &cf);
    b = nbSamp / (b * 2.0);
    y = false;
    nx = 0;
    do {
      exitg2 = 0;
      if (nx + 1 < 5) {
        if (cv1[nx] != wname[nx]) {
          exitg2 = 1;
        } else {
          nx++;
        }
      } else {
        y = true;
        exitg2 = 1;
      }
    } while (exitg2 == 0);
    if (y) {
      nx = 1;
    } else {
      nx = 0;
      do {
        exitg2 = 0;
        if (nx + 1 < 5) {
          if (cv2[nx] != wname[nx]) {
            exitg2 = 1;
          } else {
            nx++;
          }
        } else {
          y = true;
          exitg2 = 1;
        }
      } while (exitg2 == 0);
      if (y) {
        nx = 2;
      } else {
        nx = -1;
      }
    }
    switch (nx) {
    case 0:
      omegac = wavelet::internal::cwt::getFreqFromCutoffMorse(frange_idx_0, cf,
                                                              ga, be);
      break;
    case 1:
      omegac = wavelet::internal::cwt::getFreqFromCutoffBump(frange_idx_0, cf);
      break;
    case 2:
      omegac = wavelet::internal::cwt::getFreqFromCutoffAmor(frange_idx_0, cf);
      break;
    }
    N = omegac / 3.1415926535897931;
    if (b < N * rt_powd_snf(2.0, 1.0 / nv)) {
      b = N * rt_powd_snf(2.0, 1.0 / nv);
    }
    ga = rt_powd_snf(2.0, 1.0 / nv);
    emxInit_real_T(&c_y, 2, &rc_emlrtRTEI, true);
    b = std::fmax(b_log2(b / N), 1.0 / nv) * nv;
    if (std::isnan(b)) {
      i = c_y->size[0] * c_y->size[1];
      c_y->size[0] = 1;
      c_y->size[1] = 1;
      emxEnsureCapacity_real_T(c_y, i, &n_emlrtRTEI);
      c_y->data[0] = rtNaN;
    } else if (b < 0.0) {
      c_y->size[0] = 1;
      c_y->size[1] = 0;
    } else {
      i = c_y->size[0] * c_y->size[1];
      c_y->size[0] = 1;
      c_y->size[1] = static_cast<int32_T>(b) + 1;
      emxEnsureCapacity_real_T(c_y, i, &n_emlrtRTEI);
      for (i = 0; i <= static_cast<int32_T>(b); i++) {
        c_y->data[i] = i;
      }
    }
    emxInit_real_T(&d_b, 2, &sc_emlrtRTEI, true);
    i = d_b->size[0] * d_b->size[1];
    d_b->size[0] = 1;
    d_b->size[1] = c_y->size[1];
    emxEnsureCapacity_real_T(d_b, i, &kb_emlrtRTEI);
    nx = c_y->size[1];
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      d_b->data[loop_ub] = rt_powd_snf(ga, c_y->data[loop_ub]);
    }
    emxFree_real_T(&c_y);
    i = b_self->Scales->size[0] * b_self->Scales->size[1];
    b_self->Scales->size[0] = 1;
    b_self->Scales->size[1] = d_b->size[1];
    emxEnsureCapacity_real_T(b_self->Scales, i, &dc_emlrtRTEI);
    for (i = 0; i < d_b->size[1]; i++) {
      b_self->Scales->data[i] = N * d_b->data[i];
    }
    emxFree_real_T(&d_b);
  }
  wname[0] = b_self->Wavelet[0];
  wname[1] = b_self->Wavelet[1];
  wname[2] = b_self->Wavelet[2];
  wname[3] = b_self->Wavelet[3];
  emxInit_real_T(&omega, 2, &bc_emlrtRTEI, true);
  i = omega->size[0] * omega->size[1];
  omega->size[0] = 1;
  omega->size[1] = b_self->Omega->size[1];
  emxEnsureCapacity_real_T(omega, i, &bc_emlrtRTEI);
  for (i = 0; i < b_self->Omega->size[1]; i++) {
    omega->data[i] = b_self->Omega->data[i];
  }
  emxInit_real_T(&scales, 2, &cc_emlrtRTEI, true);
  i = scales->size[0] * scales->size[1];
  scales->size[0] = 1;
  scales->size[1] = b_self->Scales->size[1];
  emxEnsureCapacity_real_T(scales, i, &cc_emlrtRTEI);
  for (i = 0; i < b_self->Scales->size[1]; i++) {
    scales->data[i] = b_self->Scales->data[i];
  }
  emxInit_real_T(&somega, 2, &qc_emlrtRTEI, true);
  if (scales->size[1] == 1) {
    i = somega->size[0] * somega->size[1];
    somega->size[0] = 1;
    if (omega->size[1] == 1) {
      somega->size[1] = 1;
    } else {
      somega->size[1] = omega->size[1];
    }
    emxEnsureCapacity_real_T(somega, i, &fc_emlrtRTEI);
    if (somega->size[1] != 0) {
      nx = (omega->size[1] != 1);
      fc = somega->size[1] - 1;
      for (loop_ub = 0; loop_ub <= fc; loop_ub++) {
        somega->data[loop_ub] = scales->data[0] * omega->data[nx * loop_ub];
      }
    }
  } else {
    i = somega->size[0] * somega->size[1];
    somega->size[0] = scales->size[1];
    somega->size[1] = omega->size[1];
    emxEnsureCapacity_real_T(somega, i, &ec_emlrtRTEI);
    loop_ub = omega->size[1];
    for (i = 0; i < loop_ub; i++) {
      fc = scales->size[1];
      for (nx = 0; nx < fc; nx++) {
        somega->data[nx + somega->size[0] * i] =
            scales->data[nx] * omega->data[i];
      }
    }
  }
  emxFree_real_T(&omega);
  y = false;
  nx = 0;
  do {
    exitg2 = 0;
    if (nx + 1 < 5) {
      if (cv2[nx] != wname[nx]) {
        exitg2 = 1;
      } else {
        nx++;
      }
    } else {
      y = true;
      exitg2 = 1;
    }
  } while (exitg2 == 0);
  emxInit_real_T(&expnt, 2, &hc_emlrtRTEI, true);
  emxInit_real_T(&daughter, 2, &mc_emlrtRTEI, true);
  if (y) {
    fc = 0;
  } else {
    fc = -1;
  }
  if (fc == 0) {
    fc = 6;
    i = expnt->size[0] * expnt->size[1];
    expnt->size[0] = somega->size[0];
    expnt->size[1] = somega->size[1];
    emxEnsureCapacity_real_T(expnt, i, &hc_emlrtRTEI);
    for (i = 0; i < somega->size[0] * somega->size[1]; i++) {
      expnt->data[i] = -((somega->data[i] - 6.0) * (somega->data[i] - 6.0)) /
                       2.0 * static_cast<real_T>(somega->data[i] > 0.0);
    }
    nx = expnt->size[0] * expnt->size[1];
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      expnt->data[loop_ub] = std::exp(expnt->data[loop_ub]);
    }
    i = daughter->size[0] * daughter->size[1];
    daughter->size[0] = expnt->size[0];
    daughter->size[1] = expnt->size[1];
    emxEnsureCapacity_real_T(daughter, i, &jc_emlrtRTEI);
    for (i = 0; i < expnt->size[0] * expnt->size[1]; i++) {
      daughter->data[i] =
          2.0 * expnt->data[i] * static_cast<real_T>(somega->data[i] > 0.0);
    }
  } else {
    fc = 5;
    emxInit_real_T(&w, 2, &gc_emlrtRTEI, true);
    i = w->size[0] * w->size[1];
    w->size[0] = somega->size[0];
    w->size[1] = somega->size[1];
    emxEnsureCapacity_real_T(w, i, &gc_emlrtRTEI);
    for (i = 0; i < somega->size[0] * somega->size[1]; i++) {
      w->data[i] = (somega->data[i] - 5.0) / 0.6;
    }
    i = expnt->size[0] * expnt->size[1];
    expnt->size[0] = w->size[0];
    expnt->size[1] = w->size[1];
    emxEnsureCapacity_real_T(expnt, i, &ic_emlrtRTEI);
    for (i = 0; i < w->size[0] * w->size[1]; i++) {
      expnt->data[i] = -1.0 / (1.0 - w->data[i] * w->data[i]);
    }
    nx = expnt->size[0] * expnt->size[1];
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      expnt->data[loop_ub] = std::exp(expnt->data[loop_ub]);
    }
    nx = w->size[0] * w->size[1];
    emxInit_real_T(&d_y, 2, &tc_emlrtRTEI, true);
    i = d_y->size[0] * d_y->size[1];
    d_y->size[0] = w->size[0];
    d_y->size[1] = w->size[1];
    emxEnsureCapacity_real_T(d_y, i, &h_emlrtRTEI);
    for (loop_ub = 0; loop_ub < nx; loop_ub++) {
      d_y->data[loop_ub] = std::abs(w->data[loop_ub]);
    }
    emxFree_real_T(&w);
    if ((expnt->size[0] == d_y->size[0]) && (expnt->size[1] == d_y->size[1])) {
      i = daughter->size[0] * daughter->size[1];
      daughter->size[0] = expnt->size[0];
      daughter->size[1] = expnt->size[1];
      emxEnsureCapacity_real_T(daughter, i, &mc_emlrtRTEI);
      for (i = 0; i < expnt->size[0] * expnt->size[1]; i++) {
        daughter->data[i] =
            5.43656365691809 * expnt->data[i] *
            static_cast<real_T>(d_y->data[i] < 0.99999999999999978);
      }
    } else {
      binary_expand_op_2(daughter, expnt, d_y);
    }
    emxFree_real_T(&d_y);
    nx = daughter->size[0] * daughter->size[1] - 1;
    for (b_i = 0; b_i <= nx; b_i++) {
      if (std::isnan(daughter->data[b_i])) {
        daughter->data[b_i] = 0.0;
      }
    }
  }
  emxFree_real_T(&expnt);
  emxFree_real_T(&somega);
  ga = static_cast<real_T>(fc) / 6.2831853071795862;
  emxInit_real_T(&f, 2, &uc_emlrtRTEI, true);
  i = f->size[0] * f->size[1];
  f->size[0] = 1;
  f->size[1] = scales->size[1];
  emxEnsureCapacity_real_T(f, i, &kc_emlrtRTEI);
  for (i = 0; i < scales->size[1]; i++) {
    f->data[i] = ga / scales->data[i] * b_self->SamplingFrequency;
  }
  emxFree_real_T(&scales);
  i = b_self->PsiDFT->size[0] * b_self->PsiDFT->size[1];
  b_self->PsiDFT->size[0] = daughter->size[0];
  b_self->PsiDFT->size[1] = daughter->size[1];
  emxEnsureCapacity_real_T(b_self->PsiDFT, i, &lc_emlrtRTEI);
  for (i = 0; i < daughter->size[0] * daughter->size[1]; i++) {
    b_self->PsiDFT->data[i] = daughter->data[i];
  }
  emxFree_real_T(&daughter);
  i = b_self->WaveletCenterFrequencies->size[0];
  b_self->WaveletCenterFrequencies->size[0] = f->size[1];
  emxEnsureCapacity_real_T(b_self->WaveletCenterFrequencies, i, &nc_emlrtRTEI);
  for (i = 0; i < f->size[1]; i++) {
    b_self->WaveletCenterFrequencies->data[i] = f->data[i];
  }
  emxFree_real_T(&f);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  return b_self;
}

} // namespace coder

// End of code generation (cwtfilterbank.cu)

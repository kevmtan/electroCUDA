#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cwtfreqlimits.cu
//
// Code generation for function 'cwtfreqlimits'
//

// Include files
#include "cwtfreqlimits.h"
#include "ec_wtcc_mexutil.h"
#include "rt_nonfinite.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
namespace wavelet {
namespace internal {
namespace cwt {
real_T cwtfreqlimits(real_T N, real_T cutoff, real_T varargin_1,
                     real_T varargin_3, real_T *maxperiod, real_T *maxscale,
                     real_T *minscale, real_T *maxfreq, real_T *minperiod)
{
  real_T T;
  real_T minfreq;
  real_T omegac;
  real_T psihat_workspace_alpha;
  T = 1.0 / varargin_1;
  *maxscale = N / 2.8284271247461903;
  psihat_workspace_alpha = 2.0 * (cutoff / 100.0);
  if (psihat_workspace_alpha - 2.0 > 0.0) {
    omegac = 44.729833462074168;
  } else {
    real_T a;
    real_T fa;
    real_T fb;
    a = 6.0;
    omegac = 44.729833462074168;
    fa = psihat_workspace_alpha - 2.0;
    fb = psihat_workspace_alpha;
    if (psihat_workspace_alpha - 2.0 == 0.0) {
      omegac = 6.0;
    } else {
      real_T c;
      real_T d;
      real_T e;
      real_T fc;
      boolean_T exitg1;
      fc = psihat_workspace_alpha;
      c = 44.729833462074168;
      e = 0.0;
      d = 0.0;
      exitg1 = false;
      while ((!exitg1) && ((fb != 0.0) && (a != omegac))) {
        real_T m;
        real_T toler;
        if ((fb > 0.0) == (fc > 0.0)) {
          c = a;
          fc = fa;
          d = omegac - a;
          e = d;
        }
        if (std::abs(fc) < std::abs(fb)) {
          a = omegac;
          omegac = c;
          c = a;
          fa = fb;
          fb = fc;
          fc = fa;
        }
        m = 0.5 * (c - omegac);
        toler = 4.4408920985006262E-16 * std::fmax(std::abs(omegac), 1.0);
        if ((std::abs(m) <= toler) || (fb == 0.0)) {
          exitg1 = true;
        } else {
          if ((std::abs(e) < toler) || (std::abs(fa) <= std::abs(fb))) {
            d = m;
            e = m;
          } else {
            real_T q;
            real_T s;
            s = fb / fa;
            if (a == c) {
              fa = 2.0 * m * s;
              q = 1.0 - s;
            } else {
              real_T r;
              q = fa / fc;
              r = fb / fc;
              fa = s * (2.0 * m * q * (q - r) - (omegac - a) * (r - 1.0));
              q = (q - 1.0) * (r - 1.0) * (s - 1.0);
            }
            if (fa > 0.0) {
              q = -q;
            } else {
              fa = -fa;
            }
            if ((2.0 * fa < 3.0 * m * q - std::abs(toler * q)) &&
                (fa < std::abs(0.5 * e * q))) {
              e = d;
              d = fa / q;
            } else {
              d = m;
              e = m;
            }
          }
          a = omegac;
          fa = fb;
          if (std::abs(d) > toler) {
            omegac += d;
          } else if (omegac > c) {
            omegac -= toler;
          } else {
            omegac += toler;
          }
          fb = psihat_workspace_alpha -
               2.0 * std::exp(-((omegac - 6.0) * (omegac - 6.0)) / 2.0);
        }
      }
    }
  }
  *minscale = omegac / 3.1415926535897931;
  if (*maxscale < *minscale * rt_powd_snf(2.0, 1.0 / varargin_3)) {
    *maxscale = *minscale * rt_powd_snf(2.0, 1.0 / varargin_3);
  }
  *minperiod = *minscale * 1.0471975511965976 * T;
  *maxfreq = 1.0 / (*minscale * 1.0471975511965976) * varargin_1;
  *maxperiod = *maxscale * 1.0471975511965976 * T;
  minfreq = 1.0 / (*maxscale * 1.0471975511965976) * varargin_1;
  if ((*maxfreq > varargin_1 / 2.0) || (*minperiod < 2.0 * T)) {
    *maxfreq = varargin_1 / 2.0;
    *minperiod = 2.0 * T;
  }
  return minfreq;
}

//
//
real_T getFreqFromCutoffAmor(real_T cutoff, real_T cf)
{
  real_T a;
  real_T omegac;
  real_T psihat_workspace_alpha;
  psihat_workspace_alpha = 2.0 * cutoff;
  a = cf - cf;
  if (psihat_workspace_alpha - 2.0 * std::exp(-(a * a) / 2.0) > 0.0) {
    omegac = cf + 38.729833462074168;
  } else {
    real_T fa;
    real_T fb;
    real_T p;
    a = cf;
    omegac = cf + 38.729833462074168;
    p = cf - cf;
    fa = psihat_workspace_alpha - 2.0 * std::exp(-(p * p) / 2.0);
    p = (cf + 38.729833462074168) - cf;
    fb = psihat_workspace_alpha - 2.0 * std::exp(-(p * p) / 2.0);
    if (fa == 0.0) {
      omegac = cf;
    } else if (!(fb == 0.0)) {
      real_T c;
      real_T d;
      real_T e;
      real_T fc;
      boolean_T exitg1;
      fc = fb;
      c = cf + 38.729833462074168;
      e = 0.0;
      d = 0.0;
      exitg1 = false;
      while ((!exitg1) && ((fb != 0.0) && (a != omegac))) {
        real_T m;
        real_T toler;
        if ((fb > 0.0) == (fc > 0.0)) {
          c = a;
          fc = fa;
          d = omegac - a;
          e = d;
        }
        if (std::abs(fc) < std::abs(fb)) {
          a = omegac;
          omegac = c;
          c = a;
          fa = fb;
          fb = fc;
          fc = fa;
        }
        m = 0.5 * (c - omegac);
        toler = 4.4408920985006262E-16 * std::fmax(std::abs(omegac), 1.0);
        if ((std::abs(m) <= toler) || (fb == 0.0)) {
          exitg1 = true;
        } else {
          if ((std::abs(e) < toler) || (std::abs(fa) <= std::abs(fb))) {
            d = m;
            e = m;
          } else {
            real_T s;
            s = fb / fa;
            if (a == c) {
              p = 2.0 * m * s;
              fa = 1.0 - s;
            } else {
              real_T r;
              fa /= fc;
              r = fb / fc;
              p = s * (2.0 * m * fa * (fa - r) - (omegac - a) * (r - 1.0));
              fa = (fa - 1.0) * (r - 1.0) * (s - 1.0);
            }
            if (p > 0.0) {
              fa = -fa;
            } else {
              p = -p;
            }
            if ((2.0 * p < 3.0 * m * fa - std::abs(toler * fa)) &&
                (p < std::abs(0.5 * e * fa))) {
              e = d;
              d = p / fa;
            } else {
              d = m;
              e = m;
            }
          }
          a = omegac;
          fa = fb;
          if (std::abs(d) > toler) {
            omegac += d;
          } else if (omegac > c) {
            omegac -= toler;
          } else {
            omegac += toler;
          }
          p = omegac - cf;
          fb = psihat_workspace_alpha - 2.0 * std::exp(-(p * p) / 2.0);
        }
      }
    }
  }
  return omegac;
}

//
//
real_T getFreqFromCutoffBump(real_T cutoff, real_T cf)
{
  real_T omegac;
  int32_T exponent;
  if (cutoff < 4.94065645841247E-323) {
    real_T fa;
    real_T r;
    fa = std::abs(cf + 0.6);
    if (std::isinf(fa) || std::isnan(fa)) {
      r = rtNaN;
    } else if (fa < 4.4501477170144028E-308) {
      r = 4.94065645841247E-324;
    } else {
      std::frexp(fa, &exponent);
      r = std::ldexp(1.0, exponent - 53);
    }
    omegac = (cf + 0.6) - 10.0 * r;
  } else {
    real_T a;
    real_T epsilon;
    real_T fa;
    real_T fb;
    real_T psihat_workspace_alpha;
    psihat_workspace_alpha = 2.0 * cutoff;
    a = 4.94065645841247E-324;
    epsilon = 0.99999999999999978;
    fa = ((std::log(psihat_workspace_alpha) + 1.0) - 0.69314718055994529) - 1.0;
    fb = ((std::log(psihat_workspace_alpha) + 2.251799813685248E+15) -
          0.69314718055994529) -
         1.0;
    if (fa == 0.0) {
      epsilon = 4.94065645841247E-324;
    } else {
      real_T c;
      real_T d;
      real_T e;
      real_T fc;
      boolean_T exitg1;
      fc = fb;
      c = 0.99999999999999978;
      e = 0.0;
      d = 0.0;
      exitg1 = false;
      while ((!exitg1) && ((fb != 0.0) && (a != epsilon))) {
        real_T m;
        real_T toler;
        if ((fb > 0.0) == (fc > 0.0)) {
          c = a;
          fc = fa;
          d = epsilon - a;
          e = d;
        }
        if (std::abs(fc) < std::abs(fb)) {
          a = epsilon;
          epsilon = c;
          c = a;
          fa = fb;
          fb = fc;
          fc = fa;
        }
        m = 0.5 * (c - epsilon);
        toler = 4.4408920985006262E-16 * std::fmax(std::abs(epsilon), 1.0);
        if ((std::abs(m) <= toler) || (fb == 0.0)) {
          exitg1 = true;
        } else {
          if ((std::abs(e) < toler) || (std::abs(fa) <= std::abs(fb))) {
            d = m;
            e = m;
          } else {
            real_T q;
            real_T s;
            s = fb / fa;
            if (a == c) {
              fa = 2.0 * m * s;
              q = 1.0 - s;
            } else {
              real_T r;
              q = fa / fc;
              r = fb / fc;
              fa = s * (2.0 * m * q * (q - r) - (epsilon - a) * (r - 1.0));
              q = (q - 1.0) * (r - 1.0) * (s - 1.0);
            }
            if (fa > 0.0) {
              q = -q;
            } else {
              fa = -fa;
            }
            if ((2.0 * fa < 3.0 * m * q - std::abs(toler * q)) &&
                (fa < std::abs(0.5 * e * q))) {
              e = d;
              d = fa / q;
            } else {
              d = m;
              e = m;
            }
          }
          a = epsilon;
          fa = fb;
          if (std::abs(d) > toler) {
            epsilon += d;
          } else if (epsilon > c) {
            epsilon -= toler;
          } else {
            epsilon += toler;
          }
          fb = ((1.0 / (1.0 - epsilon * epsilon) +
                 std::log(psihat_workspace_alpha)) -
                0.69314718055994529) -
               1.0;
        }
      }
    }
    omegac = 0.6 * epsilon + cf;
  }
  return omegac;
}

//
//
real_T getFreqFromCutoffMorse(real_T cutoff, real_T cf, real_T ga, real_T be)
{
  real_T omegac;
  real_T psihat_workspace_alpha;
  real_T psihat_workspace_anorm;
  psihat_workspace_anorm =
      2.0 * std::exp(be / ga * ((std::log(ga) - std::log(be)) + 1.0));
  psihat_workspace_alpha = 2.0 * cutoff;
  omegac = rt_powd_snf(750.0, 1.0 / ga);
  if (psihat_workspace_alpha - psihat_workspace_anorm * rt_powd_snf(cf, be) *
                                   std::exp(-rt_powd_snf(cf, ga)) >=
      0.0) {
    if (!(psihat_workspace_alpha - psihat_workspace_anorm *
                                       rt_powd_snf(omegac, be) *
                                       std::exp(-rt_powd_snf(omegac, ga)) ==
          psihat_workspace_alpha - psihat_workspace_anorm *
                                       rt_powd_snf(cf, be) *
                                       std::exp(-rt_powd_snf(cf, ga)))) {
      omegac = cf;
    }
  } else {
    real_T a;
    real_T fa;
    real_T fb;
    a = cf;
    fa = psihat_workspace_alpha - psihat_workspace_anorm * rt_powd_snf(cf, be) *
                                      std::exp(-rt_powd_snf(cf, ga));
    fb = psihat_workspace_alpha - psihat_workspace_anorm *
                                      rt_powd_snf(omegac, be) *
                                      std::exp(-rt_powd_snf(omegac, ga));
    if (fa == 0.0) {
      omegac = cf;
    } else if (!(fb == 0.0)) {
      real_T c;
      real_T d;
      real_T e;
      real_T fc;
      boolean_T exitg1;
      fc = fb;
      c = omegac;
      e = 0.0;
      d = 0.0;
      exitg1 = false;
      while ((!exitg1) && ((fb != 0.0) && (a != omegac))) {
        real_T m;
        real_T toler;
        if ((fb > 0.0) == (fc > 0.0)) {
          c = a;
          fc = fa;
          d = omegac - a;
          e = d;
        }
        if (std::abs(fc) < std::abs(fb)) {
          a = omegac;
          omegac = c;
          c = a;
          fa = fb;
          fb = fc;
          fc = fa;
        }
        m = 0.5 * (c - omegac);
        toler = 4.4408920985006262E-16 * std::fmax(std::abs(omegac), 1.0);
        if ((std::abs(m) <= toler) || (fb == 0.0)) {
          exitg1 = true;
        } else {
          if ((std::abs(e) < toler) || (std::abs(fa) <= std::abs(fb))) {
            d = m;
            e = m;
          } else {
            real_T q;
            real_T s;
            s = fb / fa;
            if (a == c) {
              fa = 2.0 * m * s;
              q = 1.0 - s;
            } else {
              real_T r;
              q = fa / fc;
              r = fb / fc;
              fa = s * (2.0 * m * q * (q - r) - (omegac - a) * (r - 1.0));
              q = (q - 1.0) * (r - 1.0) * (s - 1.0);
            }
            if (fa > 0.0) {
              q = -q;
            } else {
              fa = -fa;
            }
            if ((2.0 * fa < 3.0 * m * q - std::abs(toler * q)) &&
                (fa < std::abs(0.5 * e * q))) {
              e = d;
              d = fa / q;
            } else {
              d = m;
              e = m;
            }
          }
          a = omegac;
          fa = fb;
          if (std::abs(d) > toler) {
            omegac += d;
          } else if (omegac > c) {
            omegac -= toler;
          } else {
            omegac += toler;
          }
          fb = psihat_workspace_alpha - psihat_workspace_anorm *
                                            rt_powd_snf(omegac, be) *
                                            std::exp(-rt_powd_snf(omegac, ga));
        }
      }
    }
  }
  return omegac;
}

} // namespace cwt
} // namespace internal
} // namespace wavelet
} // namespace coder

// End of code generation (cwtfreqlimits.cu)

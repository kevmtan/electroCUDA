//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_wtcc_initialize.cu
//
// Code generation for function 'ec_wtcc_initialize'
//

// Include files
#include "ec_wtcc_initialize.h"
#include "_coder_ec_wtcc_mex.h"
#include "ec_wtcc_data.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCUSOLVERUtils.hpp"

// Function Definitions
void ec_wtcc_initialize()
{
  mex_InitInfAndNan();
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "wavelet_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "signal_toolbox", 2);
  cublasEnsureInitialization(HIPBLAS_POINTER_MODE_HOST);
  cusolverEnsureInitialization();
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

// End of code generation (ec_wtcc_initialize.cu)

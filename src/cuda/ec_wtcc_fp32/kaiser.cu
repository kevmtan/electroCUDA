#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// kaiser.cu
//
// Code generation for function 'kaiser'
//

// Include files
#include "kaiser.h"
#include "casyi.h"
#include "cmlri.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "gammaln.h"
#include "rt_nonfinite.h"
#include "warning.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo nf_emlrtRTEI{
    37,                                                    // lineNo
    20,                                                    // colNo
    "kaiser",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/kaiser.m" // pName
};

static emlrtRTEInfo of_emlrtRTEI{
    39,                                                    // lineNo
    5,                                                     // colNo
    "kaiser",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/kaiser.m" // pName
};

// Function Definitions
//
//
namespace coder {
void kaiser(real_T N, emxArray_real_T *w)
{
  creal_T tmp;
  creal_T zd;
  real_T r;
  int32_T inw;
  int32_T iseven;
  int32_T nw;
  if (N == std::floor(N)) {
    nw = static_cast<int32_T>(N);
  } else {
    nw = static_cast<int32_T>(std::round(N));
    internal::warning();
  }
  iseven = w->size[0];
  w->size[0] = nw;
  emxEnsureCapacity_real_T(w, iseven, &nf_emlrtRTEI);
  if (nw <= 1) {
    iseven = w->size[0];
    w->size[0] = nw;
    emxEnsureCapacity_real_T(w, iseven, &of_emlrtRTEI);
    for (iseven = 0; iseven < nw; iseven++) {
      w->data[iseven] = 1.0;
    }
  } else {
    int64_T b;
    int32_T mid;
    iseven = 1 - (nw & 1);
    mid = (nw >> 1) + 1;
    if (computeEndIdx(static_cast<int64_T>(mid), static_cast<int64_T>(nw),
                      1L) >= 0L) {
      zd.im = 0.0;
    }
    b = computeEndIdx(static_cast<int64_T>(mid), static_cast<int64_T>(nw), 1L);
    for (int64_T k{0L}; k <= b; k++) {
      real_T hz_re;
      r = static_cast<real_T>(iseven +
                              ((static_cast<int32_T>(mid + k) - mid) << 1)) /
          (static_cast<real_T>(nw) - 1.0);
      zd.re = 5.0 * std::sqrt((1.0 - r) * (r + 1.0));
      if (std::isnan(zd.re)) {
        tmp.re = rtNaN;
        tmp.im = 0.0;
      } else {
        real_T az;
        int32_T b_nw;
        int32_T ierr;
        boolean_T guard1;
        ierr = 0;
        r = rt_hypotd_snf(zd.re, 0.0);
        if (r > 1.0737418235E+9) {
          ierr = 4;
        } else if (r > 32767.999992370605) {
          ierr = 3;
        }
        tmp.re = 0.0;
        tmp.im = 0.0;
        az = rt_hypotd_snf(zd.re, 0.0);
        guard1 = false;
        if (az <= 2.0) {
          b_nw = 0;
          r = rt_hypotd_snf(zd.re, 0.0);
          if (r == 0.0) {
            tmp.re = 1.0;
            tmp.im = 0.0;
          } else {
            real_T crsc_re;
            boolean_T iflag;
            crsc_re = 1.0;
            iflag = false;
            if (r < 2.2250738585072014E-305) {
              tmp.re = 1.0;
              tmp.im = 0.0;
            } else {
              real_T acz;
              real_T cz_re;
              hz_re = 0.5 * zd.re;
              if (r > 4.7170688552396617E-153) {
                cz_re = hz_re * hz_re;
                acz = rt_hypotd_snf(cz_re, 0.0);
              } else {
                cz_re = 0.0;
                acz = 0.0;
              }
              r = 1.0;
              gammaln(&r);
              hz_re = std::log(hz_re) * 0.0 - r;
              if (hz_re > -700.92179369444591) {
                real_T aa;
                real_T ascle;
                real_T b_atol;
                real_T coef_im;
                real_T coef_re;
                real_T s1_im;
                real_T s1_re;
                boolean_T guard2;
                ascle = 0.0;
                if (hz_re <= -664.87164553371019) {
                  iflag = true;
                  crsc_re = 2.2204460492503131E-16;
                  ascle = 1.0020841800044864E-289;
                }
                aa = std::exp(hz_re);
                if (iflag) {
                  aa /= 2.2204460492503131E-16;
                }
                coef_re = aa;
                coef_im = aa * 0.0;
                b_atol = 2.2204460492503131E-16 * acz;
                s1_re = 1.0;
                s1_im = 0.0;
                if (!(acz < 2.2204460492503131E-16)) {
                  real_T ak;
                  real_T s;
                  hz_re = 1.0;
                  r = 0.0;
                  ak = 3.0;
                  s = 1.0;
                  aa = 2.0;
                  real_T re;
                  real_T rs;
                  do {
                    rs = 1.0 / s;
                    re = hz_re * cz_re - r * 0.0;
                    r = hz_re * 0.0 + r * cz_re;
                    hz_re = rs * re;
                    r *= rs;
                    s1_re += hz_re;
                    s1_im += r;
                    s += ak;
                    ak += 2.0;
                    aa = aa * acz * rs;
                  } while (!!(aa > b_atol));
                }
                b_atol = s1_re * coef_re - s1_im * coef_im;
                r = s1_re * coef_im + s1_im * coef_re;
                guard2 = false;
                if (iflag) {
                  hz_re = std::abs(b_atol);
                  if (hz_re > r) {
                    aa = 0.0;
                  } else {
                    aa = hz_re;
                    hz_re = r;
                  }
                  if ((!(aa <= ascle)) ||
                      (!(hz_re < aa / 2.2204460492503131E-16))) {
                    guard2 = true;
                  }
                } else {
                  guard2 = true;
                }
                if (guard2) {
                  tmp.re = b_atol * crsc_re - r * 0.0;
                  tmp.im = b_atol * 0.0 + r * crsc_re;
                }
              } else {
                b_nw = 1;
                if (acz > 0.0) {
                  b_nw = -1;
                }
              }
            }
          }
          if (b_nw < 0) {
            inw = 1;
          } else {
            inw = b_nw;
          }
          if ((1 - inw != 0) && (b_nw < 0)) {
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1) {
          if (az < 21.784271729432426) {
            b_nw = cmlri(zd, 0.0, 1, &tmp);
            if (b_nw < 0) {
              if (b_nw == -2) {
                inw = -2;
              } else {
                inw = -1;
              }
            } else {
              inw = 0;
            }
          } else {
            b_nw = casyi(zd, 0.0, 1, &tmp);
            if (b_nw < 0) {
              if (b_nw == -2) {
                inw = -2;
              } else {
                inw = -1;
              }
            } else {
              inw = 0;
            }
          }
        }
        guard1 = false;
        if (inw < 0) {
          if (inw == -2) {
            tmp.re = rtNaN;
            tmp.im = 0.0;
          } else {
            ierr = 2;
            guard1 = true;
          }
        } else {
          guard1 = true;
        }
        if (guard1 && (ierr == 2)) {
          tmp.re = rtInf;
          tmp.im = 0.0;
        }
        if (zd.re > 0.0) {
          r = tmp.re;
          tmp.re = r;
          tmp.im = 0.0;
        }
      }
      if (tmp.im == 0.0) {
        r = tmp.re / 27.239871823604449;
        hz_re = 0.0;
      } else if (tmp.re == 0.0) {
        r = 0.0;
        hz_re = tmp.im / 27.239871823604449;
      } else {
        r = tmp.re / 27.239871823604449;
        hz_re = tmp.im / 27.239871823604449;
      }
      w->data[static_cast<int32_T>(mid + k) - 1] = rt_hypotd_snf(r, hz_re);
    }
    for (iseven = 0; iseven <= mid - 2; iseven++) {
      w->data[iseven] = w->data[(nw - iseven) - 1];
    }
  }
}

} // namespace coder

// End of code generation (kaiser.cu)

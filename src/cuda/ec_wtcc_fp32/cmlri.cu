#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// cmlri.cu
//
// Code generation for function 'cmlri'
//

// Include files
#include "cmlri.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_mexutil.h"
#include "gammaln.h"
#include "rt_nonfinite.h"
#include "rt_defines.h"
#include <cmath>

// Function Declarations
static real_T rt_atan2d_snf(real_T u0, real_T u1);

// Function Definitions
static real_T rt_atan2d_snf(real_T u0, real_T u1)
{
  real_T y;
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else if (std::isinf(u0) && std::isinf(u1)) {
    int32_T b;
    int32_T c;
    if (u1 > 0.0) {
      b = 1;
    } else {
      b = -1;
    }
    if (u0 > 0.0) {
      c = 1;
    } else {
      c = -1;
    }
    y = std::atan2(static_cast<real_T>(c), static_cast<real_T>(b));
  } else if (u1 == 0.0) {
    if (u0 > 0.0) {
      y = RT_PI / 2.0;
    } else if (u0 < 0.0) {
      y = -(RT_PI / 2.0);
    } else {
      y = 0.0;
    }
  } else {
    y = std::atan2(u0, u1);
  }
  return y;
}

//
//
namespace coder {
int32_T cmlri(const creal_T z, real_T fnu, int32_T kode, creal_T *y)
{
  real_T ack;
  real_T ak;
  real_T az;
  real_T bk;
  real_T ck_im;
  real_T ck_re;
  real_T flooraz;
  real_T p1_im;
  real_T p1_re;
  real_T p2_im;
  real_T p2_re;
  real_T pt_im;
  real_T pt_re;
  real_T rho2;
  real_T rz_im;
  real_T rz_re;
  real_T tst;
  int32_T i;
  int32_T icounter;
  int32_T nz;
  boolean_T earlyExit;
  boolean_T exitg1;
  nz = 0;
  az = rt_hypotd_snf(z.re, z.im);
  flooraz = std::floor(az);
  if (z.im == 0.0) {
    ck_re = (flooraz + 1.0) / z.re;
    ck_im = 0.0;
    rz_re = 2.0 / z.re;
    rz_im = 0.0;
  } else if (z.re == 0.0) {
    ck_re = 0.0;
    ck_im = -((flooraz + 1.0) / z.im);
    rz_re = 0.0;
    rz_im = -(2.0 / z.im);
  } else {
    bk = std::abs(z.re);
    ack = std::abs(z.im);
    if (bk > ack) {
      ack = z.im / z.re;
      rho2 = z.re + ack * z.im;
      ck_re = ((flooraz + 1.0) + ack * 0.0) / rho2;
      ck_im = (0.0 - ack * (flooraz + 1.0)) / rho2;
    } else if (ack == bk) {
      if (z.re > 0.0) {
        ack = 0.5;
      } else {
        ack = -0.5;
      }
      if (z.im > 0.0) {
        rho2 = 0.5;
      } else {
        rho2 = -0.5;
      }
      ck_re = ((flooraz + 1.0) * ack + 0.0 * rho2) / bk;
      ck_im = (0.0 * ack - (flooraz + 1.0) * rho2) / bk;
    } else {
      ack = z.re / z.im;
      rho2 = z.im + ack * z.re;
      ck_re = ack * (flooraz + 1.0) / rho2;
      ck_im = (ack * 0.0 - (flooraz + 1.0)) / rho2;
    }
    bk = std::abs(z.re);
    ack = std::abs(z.im);
    if (bk > ack) {
      ack = z.im / z.re;
      rho2 = z.re + ack * z.im;
      rz_re = (ack * 0.0 + 2.0) / rho2;
      rz_im = (0.0 - ack * 2.0) / rho2;
    } else if (ack == bk) {
      if (z.re > 0.0) {
        ack = 0.5;
      } else {
        ack = -0.5;
      }
      if (z.im > 0.0) {
        rho2 = 0.5;
      } else {
        rho2 = -0.5;
      }
      rz_re = (2.0 * ack + 0.0 * rho2) / bk;
      rz_im = (0.0 * ack - 2.0 * rho2) / bk;
    } else {
      ack = z.re / z.im;
      rho2 = z.im + ack * z.re;
      rz_re = ack * 2.0 / rho2;
      rz_im = (ack * 0.0 - 2.0) / rho2;
    }
  }
  p1_re = 0.0;
  p1_im = 0.0;
  p2_re = 1.0;
  p2_im = 0.0;
  ack = ((flooraz + 1.0) + 1.0) / az;
  ack += std::sqrt(ack * ack - 1.0);
  rho2 = ack * ack;
  tst = (rho2 + rho2) / ((rho2 - 1.0) * (ack - 1.0)) / 2.2204460492503131E-16;
  ak = flooraz + 1.0;
  earlyExit = true;
  icounter = 1;
  i = 1;
  exitg1 = false;
  while ((!exitg1) && (i < 81)) {
    icounter++;
    pt_re = p2_re;
    pt_im = p2_im;
    ack = ck_re * p2_re - ck_im * p2_im;
    rho2 = ck_re * p2_im + ck_im * p2_re;
    p2_re = p1_re - ack;
    p2_im = p1_im - rho2;
    p1_re = pt_re;
    p1_im = pt_im;
    ck_re += rz_re;
    ck_im += rz_im;
    if (rt_hypotd_snf(p2_re, p2_im) > tst * ak * ak) {
      earlyExit = false;
      exitg1 = true;
    } else {
      ak++;
      i++;
    }
  }
  if (earlyExit) {
    nz = -2;
  } else {
    int32_T kcounter;
    boolean_T guard1;
    kcounter = 1;
    guard1 = false;
    if (static_cast<int32_T>(flooraz) <= 0) {
      int32_T itime;
      p1_re = 0.0;
      p1_im = 0.0;
      p2_re = 1.0;
      p2_im = 0.0;
      if (z.im == 0.0) {
        ck_re = 1.0 / z.re;
        ck_im = 0.0;
      } else if (z.re == 0.0) {
        ck_re = 0.0;
        ck_im = -(1.0 / z.im);
      } else {
        bk = std::abs(z.re);
        ack = std::abs(z.im);
        if (bk > ack) {
          ack = z.im / z.re;
          rho2 = z.re + ack * z.im;
          ck_re = (ack * 0.0 + 1.0) / rho2;
          ck_im = (0.0 - ack) / rho2;
        } else if (ack == bk) {
          if (z.re > 0.0) {
            ack = 0.5;
          } else {
            ack = -0.5;
          }
          if (z.im > 0.0) {
            rho2 = 0.5;
          } else {
            rho2 = -0.5;
          }
          ck_re = (ack + 0.0 * rho2) / bk;
          ck_im = (0.0 * ack - rho2) / bk;
        } else {
          ack = z.re / z.im;
          rho2 = z.im + ack * z.re;
          ck_re = ack / rho2;
          ck_im = (ack * 0.0 - 1.0) / rho2;
        }
      }
      tst = std::sqrt(1.0 / az / 2.2204460492503131E-16);
      itime = 1;
      earlyExit = true;
      i = 1;
      exitg1 = false;
      while ((!exitg1) && (i < 81)) {
        kcounter++;
        pt_re = p2_re;
        pt_im = p2_im;
        ack = ck_re * p2_re - ck_im * p2_im;
        rho2 = ck_re * p2_im + ck_im * p2_re;
        p2_re = p1_re - ack;
        p2_im = p1_im - rho2;
        p1_re = pt_re;
        p1_im = pt_im;
        ck_re += rz_re;
        ck_im += rz_im;
        rho2 = rt_hypotd_snf(p2_re, p2_im);
        if (rho2 >= tst * ak * ak) {
          if (itime == 2) {
            earlyExit = false;
            exitg1 = true;
          } else {
            ack = rt_hypotd_snf(ck_re, ck_im);
            ack = std::fmin(ack + std::sqrt(ack * ack - 1.0),
                            rho2 / rt_hypotd_snf(pt_re, pt_im));
            tst *= std::sqrt(ack / (ack * ack - 1.0));
            itime = 2;
            i++;
          }
        } else {
          i++;
        }
      }
      if (earlyExit) {
        nz = -2;
      } else {
        guard1 = true;
      }
    } else {
      guard1 = true;
    }
    if (guard1) {
      icounter += static_cast<int32_T>(flooraz);
      if (icounter >= kcounter) {
        kcounter = icounter;
      }
      az = kcounter;
      p1_re = 0.0;
      p1_im = 0.0;
      p2_re = 1.0020841800044864E-289;
      p2_im = 0.0;
      ak = fnu + fnu;
      tst = (static_cast<real_T>(kcounter) + ak) + 1.0;
      gammaln(&tst);
      rho2 = static_cast<real_T>(kcounter) + 1.0;
      gammaln(&rho2);
      ack = ak + 1.0;
      gammaln(&ack);
      bk = std::exp((tst - rho2) - ack);
      flooraz = 0.0;
      ck_re = 0.0;
      for (i = 0; i < kcounter; i++) {
        pt_re = p2_re;
        pt_im = p2_im;
        rho2 = (az + fnu) * rz_re;
        ack = (az + fnu) * rz_im;
        tst = rho2 * p2_re - ack * p2_im;
        ack = rho2 * p2_im + ack * p2_re;
        p2_re = p1_re + tst;
        p2_im = p1_im + ack;
        p1_re = pt_re;
        p1_im = pt_im;
        ack = bk * (1.0 - ak / (az + ak));
        flooraz += (ack + bk) * pt_re;
        ck_re += (ack + bk) * pt_im;
        bk = ack;
        az--;
      }
      y->re = p2_re;
      y->im = p2_im;
      pt_re = z.re;
      pt_im = z.im;
      if (kode == 2) {
        pt_re = z.re - z.re;
        pt_im = z.im;
      }
      if (rz_im == 0.0) {
        if (rz_re < 0.0) {
          ack = std::log(std::abs(rz_re));
          tst = 3.1415926535897931;
        } else {
          ack = std::log(rz_re);
          tst = 0.0;
        }
      } else if ((std::abs(rz_re) > 8.9884656743115785E+307) ||
                 (std::abs(rz_im) > 8.9884656743115785E+307)) {
        ack = std::log(rt_hypotd_snf(rz_re / 2.0, rz_im / 2.0)) +
              0.69314718055994529;
        tst = rt_atan2d_snf(rz_im, rz_re);
      } else {
        ack = std::log(rt_hypotd_snf(rz_re, rz_im));
        tst = rt_atan2d_snf(rz_im, rz_re);
      }
      rho2 = -fnu * ack - -0.0 * tst;
      bk = -fnu * tst + -0.0 * ack;
      tst = fnu + 1.0;
      gammaln(&tst);
      ack = (rho2 + pt_re) - tst;
      tst = bk + pt_im;
      p2_re += flooraz;
      p2_im += ck_re;
      p1_re = 1.0 / rt_hypotd_snf(p2_re, p2_im);
      if (ack == 0.0) {
        ack = std::cos(tst);
        tst = std::sin(tst);
      } else if (tst == 0.0) {
        ack = std::exp(ack);
        tst = 0.0;
      } else if (std::isinf(tst) && std::isinf(ack) && (ack < 0.0)) {
        ack = 0.0;
        tst = 0.0;
      } else {
        rho2 = std::exp(ack / 2.0);
        ack = rho2 * (rho2 * std::cos(tst));
        tst = rho2 * (rho2 * std::sin(tst));
      }
      bk = ack * p1_re - tst * 0.0;
      tst = ack * 0.0 + tst * p1_re;
      rho2 = p2_re * p1_re + p2_im * 0.0;
      p2_im = p2_re * 0.0 - p2_im * p1_re;
      ack = bk * rho2 - tst * p2_im;
      rho2 = bk * p2_im + tst * rho2;
      bk = y->re * rho2 + y->im * ack;
      y->re = y->re * ack - y->im * rho2;
      y->im = bk;
    }
  }
  return nz;
}

} // namespace coder

// End of code generation (cmlri.cu)

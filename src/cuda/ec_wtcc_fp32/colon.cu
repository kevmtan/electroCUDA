#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// colon.cu
//
// Code generation for function 'colon'
//

// Include files
#include "colon.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo xd_emlrtRTEI{
    320,                                                          // lineNo
    20,                                                           // colNo
    "colon",                                                      // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/ops/colon.m" // pName
};

// Function Declarations
static __global__ void eml_float_colon_kernel29(const real_T apnd,
                                                const int32_T n, const real_T a,
                                                const int32_T b,
                                                emxArray_real_T y);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void eml_float_colon_kernel29(
    const real_T apnd, const int32_T n, const real_T a, const int32_T b,
    emxArray_real_T y)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    y.data[k + 1] = a - (static_cast<real_T>(k) + 1.0);
    y.data[(n - k) - 2] = apnd - (-(static_cast<real_T>(k) + 1.0));
  }
}

//
//
namespace coder {
void eml_float_colon(real_T a, emxArray_real_T *cpu_y,
                     boolean_T *y_outdatedOnCpu, emxArray_real_T *gpu_y,
                     boolean_T *y_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  real_T apnd;
  real_T ndbl;
  int32_T nm1d2;
  ndbl = std::floor(-(1.0 - a) + 0.5);
  apnd = a - ndbl;
  if (std::abs(1.0 - apnd) <
      4.4408920985006262E-16 * std::fmax(std::abs(a), 1.0)) {
    ndbl++;
    apnd = 1.0;
  } else if (1.0 - apnd > 0.0) {
    apnd = a - (ndbl - 1.0);
  } else {
    ndbl++;
  }
  *y_outdatedOnGpu = false;
  nm1d2 = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = 1;
  cpu_y->size[1] = static_cast<int32_T>(ndbl);
  emxEnsureCapacity_real_T(cpu_y, nm1d2, &xd_emlrtRTEI);
  if (static_cast<int32_T>(ndbl) > 0) {
    cpu_y->data[0] = a;
    *y_outdatedOnGpu = true;
    if (static_cast<int32_T>(ndbl) > 1) {
      boolean_T validLaunchParams;
      cpu_y->data[static_cast<int32_T>(ndbl) - 1] = apnd;
      nm1d2 = (static_cast<int32_T>(ndbl) - 1) / 2;
      mwGetLaunchParameters1D(computeNumIters(nm1d2 - 2), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real_T(cpu_y, gpu_y, false);
      gpuEmxMemcpyCpuToGpu_real_T(gpu_y, cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        eml_float_colon_kernel29<<<grid, block>>>(
            apnd, static_cast<int32_T>(ndbl), a, nm1d2 - 2, *gpu_y);
      }
      if (nm1d2 << 1 == static_cast<int32_T>(ndbl) - 1) {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_y, gpu_y);
        cpu_y->data[nm1d2] = (a + apnd) / 2.0;
      } else {
        gpuEmxMemcpyGpuToCpu_real_T(cpu_y, gpu_y);
        cpu_y->data[nm1d2] = a - static_cast<real_T>(nm1d2);
        cpu_y->data[nm1d2 + 1] = apnd - (-static_cast<real_T>(nm1d2));
      }
    }
  }
  *y_outdatedOnCpu = false;
}

} // namespace coder

// End of code generation (colon.cu)

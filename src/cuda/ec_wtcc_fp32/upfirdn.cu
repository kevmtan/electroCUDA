#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// upfirdn.cu
//
// Code generation for function 'upfirdn'
//

// Include files
#include "upfirdn.h"
#include "circshift.h"
#include "conv2.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo pf_emlrtRTEI{
    49,                                                     // lineNo
    5,                                                      // colNo
    "upfirdn",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/upfirdn.m" // pName
};

static emlrtRTEInfo qf_emlrtRTEI{
    47,                                                     // lineNo
    5,                                                      // colNo
    "upfirdn",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/upfirdn.m" // pName
};

static emlrtRTEInfo rf_emlrtRTEI{
    71,                                                     // lineNo
    9,                                                      // colNo
    "upfirdn",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/upfirdn.m" // pName
};

static emlrtRTEInfo sf_emlrtRTEI{
    91,                                                           // lineNo
    14,                                                           // colNo
    "downsample",                                                 // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/downsample.m" // pName
};

static emlrtRTEInfo tf_emlrtRTEI{
    83,                                                     // lineNo
    13,                                                     // colNo
    "upfirdn",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/upfirdn.m" // pName
};

static emlrtRTEInfo uf_emlrtRTEI{
    83,                                                     // lineNo
    5,                                                      // colNo
    "upfirdn",                                              // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/eml/upfirdn.m" // pName
};

// Function Declarations
static
#ifdef __HIPCC__
    __device__
#endif
        int32_T
        div_s32_device(int32_T numerator, int32_T denominator);

static __global__ void upfirdn_kernel103(const emxArray_creal32_T x,
                                         const int32_T b_x,
                                         emxArray_creal32_T xCol);

static __global__ void upfirdn_kernel32(const emxArray_real32_T x,
                                        const int32_T b_x,
                                        emxArray_real32_T xCol);

static __global__ void upfirdn_kernel33(const int32_T nx, const int32_T n,
                                        const int32_T dim, int32_T origSiz[2]);

static __global__ void upfirdn_kernel34(const emxArray_real32_T w,
                                        const int32_T nElements_dim0,
                                        const int32_T nx, const int32_T b,
                                        const int32_T c, emxArray_real32_T b_w,
                                        int32_T w_dim0);

static __global__ void upfirdn_kernel35(const emxArray_real32_T w,
                                        const int32_T nElements,
                                        emxArray_real32_T y_out);

static __global__ void upfirdn_kernel36(const emxArray_real32_T w,
                                        const int32_T nElements_dim0,
                                        const int32_T nx, const int32_T b,
                                        const int32_T c, emxArray_real32_T b_w,
                                        int32_T w_dim0);

static __global__ void upfirdn_kernel37(const emxArray_real32_T w,
                                        const int32_T nElements_dim0,
                                        const int32_T b, const int32_T c,
                                        emxArray_real32_T y_out,
                                        int32_T y_out_dim0);

static __global__ void upfirdn_kernel38(const emxArray_real32_T y_out,
                                        const int32_T b,
                                        emxArray_real32_T b_y_out,
                                        int32_T y_out_dim0);

static __global__ void upfirdn_kernel39(const emxArray_real32_T y_out,
                                        const int32_T b_y_out,
                                        emxArray_real32_T c_y_out);

static __global__ void upfirdn_kernel43(const emxArray_real32_T x,
                                        const int32_T b_x,
                                        emxArray_real32_T xCol);

static __global__ void upfirdn_kernel92(const emxArray_creal32_T x,
                                        const int32_T b_x,
                                        emxArray_creal32_T xCol);

static __global__ void upfirdn_kernel93(const int32_T nx, const int32_T n,
                                        const int32_T dim, int32_T origSiz[2]);

static __global__ void upfirdn_kernel94(const emxArray_creal32_T w,
                                        const int32_T nElements_dim0,
                                        const int32_T nx, const int32_T b,
                                        const int32_T c, emxArray_creal32_T b_w,
                                        int32_T w_dim0);

static __global__ void upfirdn_kernel95(const emxArray_creal32_T w,
                                        const int32_T nElements,
                                        emxArray_creal32_T y_out);

static __global__ void upfirdn_kernel96(const emxArray_creal32_T w,
                                        const int32_T nElements_dim0,
                                        const int32_T nx, const int32_T b,
                                        const int32_T c, emxArray_creal32_T b_w,
                                        int32_T w_dim0);

static __global__ void upfirdn_kernel97(const emxArray_creal32_T w,
                                        const int32_T nElements_dim0,
                                        const int32_T b, const int32_T c,
                                        emxArray_creal32_T y_out,
                                        int32_T y_out_dim0);

static __global__ void upfirdn_kernel98(const emxArray_creal32_T y_out,
                                        const int32_T b,
                                        emxArray_creal32_T b_y_out,
                                        int32_T y_out_dim0);

static __global__ void upfirdn_kernel99(const emxArray_creal32_T y_out,
                                        const int32_T b_y_out,
                                        emxArray_creal32_T c_y_out);

// Function Definitions
static __device__ int32_T div_s32_device(int32_T numerator, int32_T denominator)
{
  int32_T quotient;
  if (denominator == 0) {
    if (numerator >= 0) {
      quotient = MAX_int32_T;
    } else {
      quotient = MIN_int32_T;
    }
  } else {
    uint32_T b;
    uint32_T tempAbsQuotient;
    if (numerator < 0) {
      tempAbsQuotient = ~static_cast<uint32_T>(numerator) + 1U;
    } else {
      tempAbsQuotient = static_cast<uint32_T>(numerator);
    }
    if (denominator < 0) {
      b = ~static_cast<uint32_T>(denominator) + 1U;
    } else {
      b = static_cast<uint32_T>(denominator);
    }
    tempAbsQuotient /= b;
    if (static_cast<int32_T>(numerator < 0) !=
        static_cast<int32_T>(denominator < 0)) {
      quotient = -static_cast<int32_T>(tempAbsQuotient);
    } else {
      quotient = static_cast<int32_T>(tempAbsQuotient);
    }
  }
  return quotient;
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel103(
    const emxArray_creal32_T x, const int32_T b_x, emxArray_creal32_T xCol)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    xCol.data[i1] = x.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel32(
    const emxArray_real32_T x, const int32_T b_x, emxArray_real32_T xCol)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    xCol.data[i1] = x.data[i1];
  }
}

static __global__ __launch_bounds__(32, 1) void upfirdn_kernel33(
    const int32_T nx, const int32_T n, const int32_T dim, int32_T origSiz[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    origSiz[dim] = div_s32_device(n, nx) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel34(
    const emxArray_real32_T w, const int32_T nElements_dim0, const int32_T nx,
    const int32_T b, const int32_T c, emxArray_real32_T b_w, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b) + 1UL));
    b_w.data[i2 + w_dim0 * i1] = w.data[nx * i2 + nElements_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel35(
    const emxArray_real32_T w, const int32_T nElements, emxArray_real32_T y_out)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nElements);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y_out.data[i1] = w.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel36(
    const emxArray_real32_T w, const int32_T nElements_dim0, const int32_T nx,
    const int32_T b, const int32_T c, emxArray_real32_T b_w, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b) + 1UL));
    b_w.data[i2 + w_dim0 * i1] = w.data[nx * i2 + nElements_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel37(
    const emxArray_real32_T w, const int32_T nElements_dim0, const int32_T b,
    const int32_T c, emxArray_real32_T y_out, int32_T y_out_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b) + 1UL));
    y_out.data[i2 + y_out_dim0 * i1] = w.data[i2 + nElements_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel38(
    const emxArray_real32_T y_out, const int32_T b, emxArray_real32_T b_y_out,
    int32_T y_out_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y_out.data[y_out_dim0 * i1] = y_out.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel39(
    const emxArray_real32_T y_out, const int32_T b_y_out,
    emxArray_real32_T c_y_out)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y_out);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_y_out.data[i1] = y_out.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel43(
    const emxArray_real32_T x, const int32_T b_x, emxArray_real32_T xCol)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    xCol.data[i1] = x.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel92(
    const emxArray_creal32_T x, const int32_T b_x, emxArray_creal32_T xCol)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_x);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    xCol.data[i1] = x.data[i1];
  }
}

static __global__ __launch_bounds__(32, 1) void upfirdn_kernel93(
    const int32_T nx, const int32_T n, const int32_T dim, int32_T origSiz[2])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    origSiz[dim] = div_s32_device(n, nx) + 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel94(
    const emxArray_creal32_T w, const int32_T nElements_dim0, const int32_T nx,
    const int32_T b, const int32_T c, emxArray_creal32_T b_w, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b) + 1UL));
    b_w.data[i2 + w_dim0 * i1] = w.data[nx * i2 + nElements_dim0 * i1];
  }
}

static __global__
    __launch_bounds__(1024, 1) void upfirdn_kernel95(const emxArray_creal32_T w,
                                                     const int32_T nElements,
                                                     emxArray_creal32_T y_out)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(nElements);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    y_out.data[i1] = w.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel96(
    const emxArray_creal32_T w, const int32_T nElements_dim0, const int32_T nx,
    const int32_T b, const int32_T c, emxArray_creal32_T b_w, int32_T w_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b) + 1UL));
    b_w.data[i2 + w_dim0 * i1] = w.data[nx * i2 + nElements_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel97(
    const emxArray_creal32_T w, const int32_T nElements_dim0, const int32_T b,
    const int32_T c, emxArray_creal32_T y_out, int32_T y_out_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T i2;
    i2 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(i2)) /
                              (static_cast<uint64_T>(b) + 1UL));
    y_out.data[i2 + y_out_dim0 * i1] = w.data[i2 + nElements_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel98(
    const emxArray_creal32_T y_out, const int32_T b, emxArray_creal32_T b_y_out,
    int32_T y_out_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    b_y_out.data[y_out_dim0 * i1] = y_out.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void upfirdn_kernel99(
    const emxArray_creal32_T y_out, const int32_T b_y_out,
    emxArray_creal32_T c_y_out)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y_out);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c_y_out.data[i1] = y_out.data[i1];
  }
}

//
//
namespace coder {
void b_upfirdn(emxArray_creal32_T *cpu_x, emxArray_creal32_T *gpu_x,
               boolean_T *x_outdatedOnGpu, emxArray_real32_T *cpu_h,
               boolean_T *h_outdatedOnCpu, emxArray_real32_T *gpu_h,
               boolean_T *h_outdatedOnGpu, real_T varargin_1, real_T varargin_2,
               emxArray_creal32_T *cpu_y_out, boolean_T *y_out_outdatedOnCpu,
               emxArray_creal32_T *gpu_y_out, boolean_T *y_out_outdatedOnGpu)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  dim3 block;
  dim3 grid;
  emxArray_creal32_T b_gpu_w;
  emxArray_creal32_T b_gpu_y_out;
  emxArray_creal32_T b_yCol;
  emxArray_creal32_T c_gpu_w;
  emxArray_creal32_T gpu_result;
  emxArray_creal32_T gpu_w;
  emxArray_creal32_T gpu_xCol;
  emxArray_creal32_T *A;
  emxArray_creal32_T *b_cpu_w;
  emxArray_creal32_T *b_cpu_y_out;
  emxArray_creal32_T *c_cpu_w;
  emxArray_creal32_T *cpu_result;
  emxArray_creal32_T *cpu_w;
  emxArray_creal32_T *cpu_xCol;
  emxArray_creal32_T *yCol;
  emxArray_real32_T gpu_B;
  emxArray_real32_T *cpu_B;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T Ly;
  int32_T b_nElements[2];
  int32_T cpu_lshift[2];
  int32_T cpu_origSiz[2];
  int32_T(*gpu_lshift)[2];
  int32_T(*gpu_origSiz)[2];
  int32_T b_i;
  int32_T dim;
  int32_T i;
  int32_T n;
  int32_T nElements;
  int32_T nx;
  char_T unusedExpr[23];
  boolean_T B_outdatedOnCpu;
  boolean_T result_outdatedOnCpu;
  boolean_T validLaunchParams;
  boolean_T w_needsGpuEnsureCapacity;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T xIsRow;
  gpuEmxReset_creal32_T(&c_gpu_w);
  gpuEmxReset_creal32_T(&b_gpu_y_out);
  gpuEmxReset_creal32_T(&b_gpu_w);
  checkCudaError(mwCudaMalloc(&gpu_lshift, 8UL), __FILE__, __LINE__);
  gpuEmxReset_creal32_T(&gpu_result);
  gpuEmxReset_real32_T(&gpu_B);
  gpuEmxReset_creal32_T(&gpu_w);
  checkCudaError(mwCudaMalloc(&gpu_origSiz, 8UL), __FILE__, __LINE__);
  gpuEmxReset_creal32_T(&gpu_xCol);
  w_outdatedOnCpu = false;
  w_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  xIsRow = (cpu_x->size[0] == 1);
  emxInit_creal32_T(&cpu_xCol, 2, &qf_emlrtRTEI, true);
  if (xIsRow) {
    i = cpu_xCol->size[0] * cpu_xCol->size[1];
    cpu_xCol->size[0] = cpu_x->size[0] * cpu_x->size[1];
    cpu_xCol->size[1] = 1;
    emxEnsureCapacity_creal32_T(cpu_xCol, i, &qf_emlrtRTEI);
    i = cpu_x->size[0] * cpu_x->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    gpuEmxEnsureCapacity_creal32_T(cpu_xCol, &gpu_xCol, true);
    if (*x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel92<<<grid, block>>>(*gpu_x, i, gpu_xCol);
    }
    result_outdatedOnCpu = true;
  } else {
    i = cpu_xCol->size[0] * cpu_xCol->size[1];
    cpu_xCol->size[0] = cpu_x->size[0];
    cpu_xCol->size[1] = cpu_x->size[1];
    emxEnsureCapacity_creal32_T(cpu_xCol, i, &pf_emlrtRTEI);
    i = cpu_x->size[0] * cpu_x->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    gpuEmxEnsureCapacity_creal32_T(cpu_xCol, &gpu_xCol, true);
    if (*x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel103<<<grid, block>>>(*gpu_x, i, gpu_xCol);
    }
    result_outdatedOnCpu = true;
  }
  if (!(varargin_1 - 1.0 >= 0.0)) {
    y = nullptr;
    m = emlrtCreateCharArray(2, &iv[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
    emlrtAssign(&y, m);
    b_y = nullptr;
    m1 = emlrtCreateDoubleScalar(varargin_1 - 1.0);
    emlrtAssign(&b_y, m1);
    emlrt_marshallIn(b_sprintf(y, b_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  nx = cpu_xCol->size[0] * cpu_xCol->size[1];
  if ((cpu_xCol->size[0] == 1) && (cpu_xCol->size[1] == 1)) {
    dim = 0;
  } else {
    dim = 1;
    if (cpu_xCol->size[0] != 1) {
      dim = 0;
    }
  }
  for (i = 0; i < 2; i++) {
    cpu_origSiz[i] = cpu_xCol->size[i];
  }
  nElements = cpu_xCol->size[0] * cpu_xCol->size[1];
  emxInit_creal32_T(&yCol, 1, &db_emlrtRTEI, true);
  i = yCol->size[0];
  yCol->size[0] = nx * static_cast<int32_T>(varargin_1);
  emxEnsureCapacity_creal32_T(yCol, i, &t_emlrtRTEI);
  for (i = 0; i < nx * static_cast<int32_T>(varargin_1); i++) {
    yCol->data[i].re = 0.0F;
    yCol->data[i].im = 0.0F;
  }
  if ((static_cast<int32_T>(varargin_1) == 0) ||
      (((static_cast<int32_T>(varargin_1) > 0) &&
        (nElements * static_cast<int32_T>(varargin_1) < 1)) ||
       ((static_cast<int32_T>(varargin_1) < 0) &&
        (nElements * static_cast<int32_T>(varargin_1) > 1)))) {
    b_i = 1;
    nElements = 0;
  } else {
    b_i = static_cast<int32_T>(varargin_1);
    nElements = nx * static_cast<int32_T>(varargin_1);
  }
  nElements = div_s32(nElements - 1, b_i) + 1;
  for (i = 0; i < nElements; i++) {
    if (result_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xCol, &gpu_xCol);
    }
    result_outdatedOnCpu = false;
    yCol->data[b_i * i] = cpu_xCol->data[i];
  }
  cpu_origSiz[dim] = cpu_xCol->size[dim] * static_cast<int32_T>(varargin_1);
  emxInit_creal32_T(&cpu_w, 2, &v_emlrtRTEI, true);
  emxInit_creal32_T(&A, 1, &v_emlrtRTEI, true);
  emxInit_real32_T(&cpu_B, 1, &rf_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_result, 1, &v_emlrtRTEI, true);
  b_yCol = *yCol;
  b_nElements[0] = cpu_origSiz[0];
  b_nElements[1] = cpu_origSiz[1];
  b_yCol.size = &b_nElements[0];
  b_yCol.numDimensions = 2;
  c_conv2(&b_yCol, cpu_h, h_outdatedOnCpu, gpu_h, h_outdatedOnGpu, cpu_w,
          &w_outdatedOnCpu, &gpu_w, &w_outdatedOnGpu);
  w_needsGpuEnsureCapacity = true;
  emxFree_creal32_T(&cpu_result);
  emxFree_real32_T(&cpu_B);
  emxFree_creal32_T(&A);
  emxFree_creal32_T(&yCol);
  if (!(varargin_2 - 1.0 >= 0.0)) {
    c_y = nullptr;
    m2 = emlrtCreateCharArray(2, &iv1[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
    emlrtAssign(&c_y, m2);
    d_y = nullptr;
    m3 = emlrtCreateDoubleScalar(varargin_2 - 1.0);
    emlrtAssign(&d_y, m3);
    emlrt_marshallIn(b_sprintf(c_y, d_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  dim = 1;
  if (cpu_w->size[0] != 1) {
    dim = 0;
  }
  result_outdatedOnCpu = false;
  w_outdatedOnCpu = false;
  B_outdatedOnCpu = false;
  for (i = 0; i < 2; i++) {
    cpu_origSiz[i] = cpu_w->size[i];
    result_outdatedOnCpu = true;
    cpu_lshift[i] = cpu_w->size[i];
    B_outdatedOnCpu = true;
  }
  circshift(cpu_lshift, &w_outdatedOnCpu, *gpu_lshift, &B_outdatedOnCpu, -dim);
  if (w_outdatedOnCpu) {
    checkCudaError(
        hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  if ((varargin_2 == 0.0) || ((varargin_2 > 0.0) && (cpu_lshift[0] < 1))) {
    nx = 1;
    n = -1;
  } else {
    nx = static_cast<int32_T>(varargin_2);
    n = cpu_lshift[0] - 1;
  }
  if (result_outdatedOnCpu) {
    checkCudaError(
        hipMemcpy(*gpu_origSiz, cpu_origSiz, 8UL, hipMemcpyHostToDevice),
        __FILE__, __LINE__);
  }
  upfirdn_kernel93<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(nx, n, dim,
                                                            *gpu_origSiz);
  emxInit_creal32_T(&b_cpu_w, 2, &sf_emlrtRTEI, true);
  nElements = b_cpu_w->size[0] * b_cpu_w->size[1];
  b_cpu_w->size[0] = div_s32(n, nx) + 1;
  b_cpu_w->size[1] = cpu_lshift[1];
  emxEnsureCapacity_creal32_T(b_cpu_w, nElements, &sf_emlrtRTEI);
  i = n / nx;
  mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, i), &grid, &block,
                          2147483647U);
  if (w_needsGpuEnsureCapacity) {
    gpuEmxEnsureCapacity_creal32_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
  }
  gpuEmxEnsureCapacity_creal32_T(b_cpu_w, &b_gpu_w, true);
  if (w_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_w, cpu_w);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    upfirdn_kernel94<<<grid, block>>>(gpu_w, cpu_lshift[0], nx, i,
                                      cpu_lshift[1] - 1, b_gpu_w,
                                      b_cpu_w->size[0U]);
  }
  checkCudaError(
      hipMemcpy(cpu_origSiz, *gpu_origSiz, 8UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  i = cpu_y_out->size[0] * cpu_y_out->size[1];
  cpu_y_out->size[0] = cpu_origSiz[0];
  cpu_y_out->size[1] = cpu_origSiz[1];
  emxEnsureCapacity_creal32_T(cpu_y_out, i, &v_emlrtRTEI);
  nElements = cpu_origSiz[0] * cpu_origSiz[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(nElements), &grid, &block,
                          2147483647U);
  gpuEmxEnsureCapacity_creal32_T(cpu_y_out, gpu_y_out, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    upfirdn_kernel95<<<grid, block>>>(b_gpu_w, nElements, *gpu_y_out);
  }
  emxFree_creal32_T(&b_cpu_w);
  Ly = std::ceil(((static_cast<real_T>(cpu_xCol->size[0]) - 1.0) * varargin_1 +
                  static_cast<real_T>(cpu_h->size[0])) /
                 varargin_2);
  emxFree_creal32_T(&cpu_xCol);
  if (!(cpu_origSiz[0] < Ly)) {
    if (Ly < 1.0) {
      b_i = 0;
    } else {
      b_i = static_cast<int32_T>(Ly);
    }
    emxInit_creal32_T(&c_cpu_w, 2, &sf_emlrtRTEI, true);
    nElements = c_cpu_w->size[0] * c_cpu_w->size[1];
    c_cpu_w->size[0] = div_s32(n, nx) + 1;
    c_cpu_w->size[1] = cpu_lshift[1];
    emxEnsureCapacity_creal32_T(c_cpu_w, nElements, &sf_emlrtRTEI);
    i = n / nx;
    mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, i), &grid,
                            &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(c_cpu_w, &c_gpu_w, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel96<<<grid, block>>>(gpu_w, cpu_lshift[0], nx, i,
                                        cpu_lshift[1] - 1, c_gpu_w,
                                        c_cpu_w->size[0U]);
    }
    nElements = cpu_y_out->size[0] * cpu_y_out->size[1];
    cpu_y_out->size[0] = b_i;
    cpu_y_out->size[1] = cpu_origSiz[1];
    emxEnsureCapacity_creal32_T(cpu_y_out, nElements, &w_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(cpu_origSiz[1] - 1, b_i - 1), &grid,
                            &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_y_out, gpu_y_out, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel97<<<grid, block>>>(c_gpu_w, cpu_origSiz[0], b_i - 1,
                                        cpu_origSiz[1] - 1, *gpu_y_out,
                                        cpu_y_out->size[0U]);
    }
    emxFree_creal32_T(&c_cpu_w);
  }
  emxFree_creal32_T(&cpu_w);
  if (xIsRow) {
    nElements = cpu_y_out->size[0] * cpu_y_out->size[1];
    emxInit_creal32_T(&b_cpu_y_out, 2, &tf_emlrtRTEI, true);
    i = b_cpu_y_out->size[0] * b_cpu_y_out->size[1];
    b_cpu_y_out->size[0] = 1;
    b_cpu_y_out->size[1] = nElements;
    emxEnsureCapacity_creal32_T(b_cpu_y_out, i, &tf_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nElements - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_creal32_T(b_cpu_y_out, &b_gpu_y_out, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel98<<<grid, block>>>(*gpu_y_out, nElements - 1, b_gpu_y_out,
                                        b_cpu_y_out->size[0U]);
    }
    i = cpu_y_out->size[0] * cpu_y_out->size[1];
    cpu_y_out->size[0] = 1;
    cpu_y_out->size[1] = b_cpu_y_out->size[1];
    emxEnsureCapacity_creal32_T(cpu_y_out, i, &uf_emlrtRTEI);
    i = b_cpu_y_out->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_y_out, gpu_y_out, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel99<<<grid, block>>>(b_gpu_y_out, i, *gpu_y_out);
    }
    emxFree_creal32_T(&b_cpu_y_out);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_creal32_T(&gpu_xCol);
  checkCudaError(mwCudaFree(*gpu_origSiz), __FILE__, __LINE__);
  gpuEmxFree_creal32_T(&gpu_w);
  gpuEmxFree_real32_T(&gpu_B);
  gpuEmxFree_creal32_T(&gpu_result);
  checkCudaError(mwCudaFree(*gpu_lshift), __FILE__, __LINE__);
  gpuEmxFree_creal32_T(&b_gpu_w);
  gpuEmxFree_creal32_T(&b_gpu_y_out);
  gpuEmxFree_creal32_T(&c_gpu_w);
  *x_outdatedOnGpu = false;
  *y_out_outdatedOnCpu = true;
  *y_out_outdatedOnGpu = false;
}

//
//
void upfirdn(emxArray_real32_T *cpu_x, emxArray_real32_T *gpu_x,
             boolean_T *x_outdatedOnGpu, emxArray_real32_T *cpu_h,
             boolean_T *h_outdatedOnCpu, emxArray_real32_T *gpu_h,
             boolean_T *h_outdatedOnGpu, real_T varargin_1, real_T varargin_2,
             emxArray_real32_T *cpu_y_out, boolean_T *y_out_outdatedOnCpu,
             emxArray_real32_T *gpu_y_out, boolean_T *y_out_outdatedOnGpu)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  dim3 block;
  dim3 grid;
  emxArray_real32_T b_gpu_w;
  emxArray_real32_T b_gpu_y_out;
  emxArray_real32_T b_yCol;
  emxArray_real32_T c_gpu_w;
  emxArray_real32_T gpu_B;
  emxArray_real32_T gpu_result;
  emxArray_real32_T gpu_w;
  emxArray_real32_T gpu_xCol;
  emxArray_real32_T *A;
  emxArray_real32_T *b_cpu_w;
  emxArray_real32_T *b_cpu_y_out;
  emxArray_real32_T *c_cpu_w;
  emxArray_real32_T *cpu_B;
  emxArray_real32_T *cpu_result;
  emxArray_real32_T *cpu_w;
  emxArray_real32_T *cpu_xCol;
  emxArray_real32_T *yCol;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  const mxArray *y;
  real_T Ly;
  int32_T b_nElements[2];
  int32_T cpu_lshift[2];
  int32_T cpu_origSiz[2];
  int32_T(*gpu_lshift)[2];
  int32_T(*gpu_origSiz)[2];
  int32_T b_i;
  int32_T dim;
  int32_T i;
  int32_T n;
  int32_T nElements;
  int32_T nx;
  char_T unusedExpr[23];
  boolean_T B_outdatedOnCpu;
  boolean_T result_outdatedOnCpu;
  boolean_T validLaunchParams;
  boolean_T w_outdatedOnCpu;
  boolean_T w_outdatedOnGpu;
  boolean_T xCol_outdatedOnCpu;
  boolean_T xIsRow;
  gpuEmxReset_real32_T(&c_gpu_w);
  gpuEmxReset_real32_T(&b_gpu_y_out);
  gpuEmxReset_real32_T(&b_gpu_w);
  checkCudaError(mwCudaMalloc(&gpu_lshift, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real32_T(&gpu_result);
  gpuEmxReset_real32_T(&gpu_B);
  gpuEmxReset_real32_T(&gpu_w);
  checkCudaError(mwCudaMalloc(&gpu_origSiz, 8UL), __FILE__, __LINE__);
  gpuEmxReset_real32_T(&gpu_xCol);
  w_outdatedOnCpu = false;
  w_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  xIsRow = (cpu_x->size[0] == 1);
  emxInit_real32_T(&cpu_xCol, 2, &qf_emlrtRTEI, true);
  if (xIsRow) {
    i = cpu_xCol->size[0] * cpu_xCol->size[1];
    cpu_xCol->size[0] = cpu_x->size[0] * cpu_x->size[1];
    cpu_xCol->size[1] = 1;
    emxEnsureCapacity_real32_T(cpu_xCol, i, &qf_emlrtRTEI);
    i = cpu_x->size[0] * cpu_x->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    gpuEmxEnsureCapacity_real32_T(cpu_xCol, &gpu_xCol, true);
    if (*x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel32<<<grid, block>>>(*gpu_x, i, gpu_xCol);
    }
    xCol_outdatedOnCpu = true;
  } else {
    i = cpu_xCol->size[0] * cpu_xCol->size[1];
    cpu_xCol->size[0] = cpu_x->size[0];
    cpu_xCol->size[1] = cpu_x->size[1];
    emxEnsureCapacity_real32_T(cpu_xCol, i, &pf_emlrtRTEI);
    i = cpu_x->size[0] * cpu_x->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_x, gpu_x, !*x_outdatedOnGpu);
    gpuEmxEnsureCapacity_real32_T(cpu_xCol, &gpu_xCol, true);
    if (*x_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel43<<<grid, block>>>(*gpu_x, i, gpu_xCol);
    }
    xCol_outdatedOnCpu = true;
  }
  if (!(varargin_1 - 1.0 >= 0.0)) {
    y = nullptr;
    m = emlrtCreateCharArray(2, &iv[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
    emlrtAssign(&y, m);
    b_y = nullptr;
    m1 = emlrtCreateDoubleScalar(varargin_1 - 1.0);
    emlrtAssign(&b_y, m1);
    emlrt_marshallIn(b_sprintf(y, b_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  nx = cpu_xCol->size[0] * cpu_xCol->size[1];
  if ((cpu_xCol->size[0] == 1) && (cpu_xCol->size[1] == 1)) {
    dim = 0;
  } else {
    dim = 1;
    if (cpu_xCol->size[0] != 1) {
      dim = 0;
    }
  }
  for (i = 0; i < 2; i++) {
    cpu_origSiz[i] = cpu_xCol->size[i];
  }
  nElements = cpu_xCol->size[0] * cpu_xCol->size[1];
  emxInit_real32_T(&yCol, 1, &db_emlrtRTEI, true);
  i = yCol->size[0];
  yCol->size[0] = nx * static_cast<int32_T>(varargin_1);
  emxEnsureCapacity_real32_T(yCol, i, &t_emlrtRTEI);
  for (i = 0; i < nx * static_cast<int32_T>(varargin_1); i++) {
    yCol->data[i] = 0.0F;
  }
  if ((static_cast<int32_T>(varargin_1) == 0) ||
      (((static_cast<int32_T>(varargin_1) > 0) &&
        (nElements * static_cast<int32_T>(varargin_1) < 1)) ||
       ((static_cast<int32_T>(varargin_1) < 0) &&
        (nElements * static_cast<int32_T>(varargin_1) > 1)))) {
    b_i = 1;
    nElements = 0;
  } else {
    b_i = static_cast<int32_T>(varargin_1);
    nElements = nx * static_cast<int32_T>(varargin_1);
  }
  nElements = div_s32(nElements - 1, b_i) + 1;
  for (i = 0; i < nElements; i++) {
    if (xCol_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_xCol, &gpu_xCol);
    }
    xCol_outdatedOnCpu = false;
    yCol->data[b_i * i] = cpu_xCol->data[i];
  }
  cpu_origSiz[dim] = cpu_xCol->size[dim] * static_cast<int32_T>(varargin_1);
  emxInit_real32_T(&cpu_w, 2, &v_emlrtRTEI, true);
  emxInit_real32_T(&A, 1, &v_emlrtRTEI, true);
  emxInit_real32_T(&cpu_B, 1, &rf_emlrtRTEI, true);
  emxInit_real32_T(&cpu_result, 1, &v_emlrtRTEI, true);
  b_yCol = *yCol;
  b_nElements[0] = cpu_origSiz[0];
  b_nElements[1] = cpu_origSiz[1];
  b_yCol.size = &b_nElements[0];
  b_yCol.numDimensions = 2;
  conv2(&b_yCol, cpu_h, h_outdatedOnCpu, gpu_h, h_outdatedOnGpu, cpu_w,
        &w_outdatedOnCpu, &gpu_w, &w_outdatedOnGpu);
  result_outdatedOnCpu = true;
  emxFree_real32_T(&cpu_result);
  emxFree_real32_T(&cpu_B);
  emxFree_real32_T(&A);
  emxFree_real32_T(&yCol);
  if (!(varargin_2 - 1.0 >= 0.0)) {
    c_y = nullptr;
    m2 = emlrtCreateCharArray(2, &iv1[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
    emlrtAssign(&c_y, m2);
    d_y = nullptr;
    m3 = emlrtCreateDoubleScalar(varargin_2 - 1.0);
    emlrtAssign(&d_y, m3);
    emlrt_marshallIn(b_sprintf(c_y, d_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  dim = 1;
  if (cpu_w->size[0] != 1) {
    dim = 0;
  }
  xCol_outdatedOnCpu = false;
  w_outdatedOnCpu = false;
  B_outdatedOnCpu = false;
  for (i = 0; i < 2; i++) {
    cpu_origSiz[i] = cpu_w->size[i];
    xCol_outdatedOnCpu = true;
    cpu_lshift[i] = cpu_w->size[i];
    B_outdatedOnCpu = true;
  }
  circshift(cpu_lshift, &w_outdatedOnCpu, *gpu_lshift, &B_outdatedOnCpu, -dim);
  if (w_outdatedOnCpu) {
    checkCudaError(
        hipMemcpy(cpu_lshift, *gpu_lshift, 8UL, hipMemcpyDeviceToHost),
        __FILE__, __LINE__);
  }
  if ((varargin_2 == 0.0) || ((varargin_2 > 0.0) && (cpu_lshift[0] < 1))) {
    nx = 1;
    n = -1;
  } else {
    nx = static_cast<int32_T>(varargin_2);
    n = cpu_lshift[0] - 1;
  }
  if (xCol_outdatedOnCpu) {
    checkCudaError(
        hipMemcpy(*gpu_origSiz, cpu_origSiz, 8UL, hipMemcpyHostToDevice),
        __FILE__, __LINE__);
  }
  upfirdn_kernel33<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(nx, n, dim,
                                                            *gpu_origSiz);
  emxInit_real32_T(&b_cpu_w, 2, &sf_emlrtRTEI, true);
  nElements = b_cpu_w->size[0] * b_cpu_w->size[1];
  b_cpu_w->size[0] = div_s32(n, nx) + 1;
  b_cpu_w->size[1] = cpu_lshift[1];
  emxEnsureCapacity_real32_T(b_cpu_w, nElements, &sf_emlrtRTEI);
  i = n / nx;
  mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, i), &grid, &block,
                          2147483647U);
  if (result_outdatedOnCpu) {
    gpuEmxEnsureCapacity_real32_T(cpu_w, &gpu_w, !w_outdatedOnGpu);
  }
  gpuEmxEnsureCapacity_real32_T(b_cpu_w, &b_gpu_w, true);
  if (w_outdatedOnGpu) {
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_w, cpu_w);
  }
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    upfirdn_kernel34<<<grid, block>>>(gpu_w, cpu_lshift[0], nx, i,
                                      cpu_lshift[1] - 1, b_gpu_w,
                                      b_cpu_w->size[0U]);
  }
  checkCudaError(
      hipMemcpy(cpu_origSiz, *gpu_origSiz, 8UL, hipMemcpyDeviceToHost),
      __FILE__, __LINE__);
  i = cpu_y_out->size[0] * cpu_y_out->size[1];
  cpu_y_out->size[0] = cpu_origSiz[0];
  cpu_y_out->size[1] = cpu_origSiz[1];
  emxEnsureCapacity_real32_T(cpu_y_out, i, &v_emlrtRTEI);
  nElements = cpu_origSiz[0] * cpu_origSiz[1] - 1;
  mwGetLaunchParameters1D(computeNumIters(nElements), &grid, &block,
                          2147483647U);
  gpuEmxEnsureCapacity_real32_T(cpu_y_out, gpu_y_out, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    upfirdn_kernel35<<<grid, block>>>(b_gpu_w, nElements, *gpu_y_out);
  }
  emxFree_real32_T(&b_cpu_w);
  Ly = std::ceil(((static_cast<real_T>(cpu_xCol->size[0]) - 1.0) * varargin_1 +
                  static_cast<real_T>(cpu_h->size[0])) /
                 varargin_2);
  emxFree_real32_T(&cpu_xCol);
  if (!(cpu_origSiz[0] < Ly)) {
    if (Ly < 1.0) {
      b_i = 0;
    } else {
      b_i = static_cast<int32_T>(Ly);
    }
    emxInit_real32_T(&c_cpu_w, 2, &sf_emlrtRTEI, true);
    nElements = c_cpu_w->size[0] * c_cpu_w->size[1];
    c_cpu_w->size[0] = div_s32(n, nx) + 1;
    c_cpu_w->size[1] = cpu_lshift[1];
    emxEnsureCapacity_real32_T(c_cpu_w, nElements, &sf_emlrtRTEI);
    i = n / nx;
    mwGetLaunchParameters1D(computeNumIters(cpu_lshift[1] - 1, i), &grid,
                            &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(c_cpu_w, &c_gpu_w, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel36<<<grid, block>>>(gpu_w, cpu_lshift[0], nx, i,
                                        cpu_lshift[1] - 1, c_gpu_w,
                                        c_cpu_w->size[0U]);
    }
    nElements = cpu_y_out->size[0] * cpu_y_out->size[1];
    cpu_y_out->size[0] = b_i;
    cpu_y_out->size[1] = cpu_origSiz[1];
    emxEnsureCapacity_real32_T(cpu_y_out, nElements, &w_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(cpu_origSiz[1] - 1, b_i - 1), &grid,
                            &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_y_out, gpu_y_out, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel37<<<grid, block>>>(c_gpu_w, cpu_origSiz[0], b_i - 1,
                                        cpu_origSiz[1] - 1, *gpu_y_out,
                                        cpu_y_out->size[0U]);
    }
    emxFree_real32_T(&c_cpu_w);
  }
  emxFree_real32_T(&cpu_w);
  if (xIsRow) {
    nElements = cpu_y_out->size[0] * cpu_y_out->size[1];
    emxInit_real32_T(&b_cpu_y_out, 2, &tf_emlrtRTEI, true);
    i = b_cpu_y_out->size[0] * b_cpu_y_out->size[1];
    b_cpu_y_out->size[0] = 1;
    b_cpu_y_out->size[1] = nElements;
    emxEnsureCapacity_real32_T(b_cpu_y_out, i, &tf_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(nElements - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(b_cpu_y_out, &b_gpu_y_out, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel38<<<grid, block>>>(*gpu_y_out, nElements - 1, b_gpu_y_out,
                                        b_cpu_y_out->size[0U]);
    }
    i = cpu_y_out->size[0] * cpu_y_out->size[1];
    cpu_y_out->size[0] = 1;
    cpu_y_out->size[1] = b_cpu_y_out->size[1];
    emxEnsureCapacity_real32_T(cpu_y_out, i, &uf_emlrtRTEI);
    i = b_cpu_y_out->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_y_out, gpu_y_out, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      upfirdn_kernel39<<<grid, block>>>(b_gpu_y_out, i, *gpu_y_out);
    }
    emxFree_real32_T(&b_cpu_y_out);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_xCol);
  checkCudaError(mwCudaFree(*gpu_origSiz), __FILE__, __LINE__);
  gpuEmxFree_real32_T(&gpu_w);
  gpuEmxFree_real32_T(&gpu_B);
  gpuEmxFree_real32_T(&gpu_result);
  checkCudaError(mwCudaFree(*gpu_lshift), __FILE__, __LINE__);
  gpuEmxFree_real32_T(&b_gpu_w);
  gpuEmxFree_real32_T(&b_gpu_y_out);
  gpuEmxFree_real32_T(&c_gpu_w);
  *x_outdatedOnGpu = false;
  *y_out_outdatedOnCpu = true;
  *y_out_outdatedOnGpu = false;
}

} // namespace coder

// End of code generation (upfirdn.cu)

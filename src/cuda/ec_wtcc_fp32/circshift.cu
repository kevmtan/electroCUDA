#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// circshift.cu
//
// Code generation for function 'circshift'
//

// Include files
#include "circshift.h"
#include "ec_wtcc_mexutil.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"

// Function Declarations
static __global__ void circshift_kernel205(const int32_T shiftLen,
                                           const int32_T b, int32_T a[2]);

// Function Definitions
static __global__ __launch_bounds__(1024, 1) void circshift_kernel205(
    const int32_T shiftLen, const int32_T b, int32_T a[2])
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = a[0];
    a[0] = a[1 - shiftLen];
    a[1 - shiftLen] = i2;
  }
}

//
//
namespace coder {
void circshift(int32_T cpu_a[2], boolean_T *a_outdatedOnCpu, int32_T gpu_a[2],
               boolean_T *a_outdatedOnGpu, int32_T p)
{
  dim3 block;
  dim3 grid;
  int32_T shiftLen;
  shiftLen = p;
  if (p < 0) {
    shiftLen = 1;
  }
  if (shiftLen < 2 - shiftLen) {
    boolean_T validLaunchParams;
    mwGetLaunchParameters1D(
        computeNumIters(
            static_cast<int32_T>(static_cast<real_T>(2 - shiftLen) / 2.0) - 1),
        &grid, &block, 2147483647U);
    if (*a_outdatedOnGpu) {
      checkCudaError(hipMemcpy(gpu_a, cpu_a, 8UL, hipMemcpyHostToDevice),
                     __FILE__, __LINE__);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      circshift_kernel205<<<grid, block>>>(
          shiftLen,
          static_cast<int32_T>(static_cast<real_T>(2 - shiftLen) / 2.0) - 1,
          gpu_a);
    }
    *a_outdatedOnCpu = true;
  }
  if (*a_outdatedOnCpu) {
    checkCudaError(hipMemcpy(cpu_a, gpu_a, 8UL, hipMemcpyDeviceToHost),
                   __FILE__, __LINE__);
  }
  shiftLen = cpu_a[0];
  cpu_a[0] = cpu_a[1];
  cpu_a[1] = shiftLen;
  *a_outdatedOnCpu = false;
  *a_outdatedOnGpu = true;
}

} // namespace coder

// End of code generation (circshift.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wcoherence.cu
//
// Code generation for function 'wcoherence'
//

// Include files
#include "wcoherence.h"
#include "bsxfun.h"
#include "cwtfilterbank.h"
#include "cwtfreqlimits.h"
#include "div.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "fft.h"
#include "ifft.h"
#include "log2.h"
#include "rt_nonfinite.h"
#include "smoothCFS.h"
#include "wt.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo fb_emlrtRTEI{
    187,                                                            // lineNo
    1,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo gb_emlrtRTEI{
    102,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo mb_emlrtRTEI{
    219,                                                            // lineNo
    1,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo ob_emlrtRTEI{
    170,                                                            // lineNo
    5,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo pb_emlrtRTEI{
    1,                                                              // lineNo
    37,                                                             // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo rb_emlrtRTEI{
    161,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo sb_emlrtRTEI{
    216,                                                            // lineNo
    39,                                                             // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo tb_emlrtRTEI{
    217,                                                            // lineNo
    39,                                                             // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo ub_emlrtRTEI{
    220,                                                            // lineNo
    7,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

// Function Declarations
static void binary_expand_op_1(emxArray_creal32_T *in1,
                               const emxArray_creal32_T *in2,
                               const emxArray_creal32_T *in3);

static real32_T rt_hypotf_snf(real32_T u0, real32_T u1);

static __global__ void wcoherence_kernel26(const int32_T i, const int32_T b,
                                           emxArray_real32_T x);

// Function Definitions
static void binary_expand_op_1(emxArray_creal32_T *in1,
                               const emxArray_creal32_T *in2,
                               const emxArray_creal32_T *in3)
{
  int32_T aux_0_1;
  int32_T aux_1_1;
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_0_1;
  int32_T stride_1_0;
  int32_T stride_1_1;
  i = in1->size[0] * in1->size[1];
  if (in3->size[0] == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3->size[0];
  }
  if (in3->size[1] == 1) {
    in1->size[1] = in2->size[1];
  } else {
    in1->size[1] = in3->size[1];
  }
  emxEnsureCapacity_creal32_T(in1, i, &mb_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_0 = (in3->size[0] != 1);
  stride_1_1 = (in3->size[1] != 1);
  aux_0_1 = 0;
  aux_1_1 = 0;
  if (in3->size[1] == 1) {
    b = in2->size[1];
  } else {
    b = in3->size[1];
  }
  for (i = 0; i < b; i++) {
    int32_T c;
    if (in3->size[0] == 1) {
      c = in2->size[0];
    } else {
      c = in3->size[0];
    }
    for (int32_T i1{0}; i1 < c; i1++) {
      real32_T in3_im;
      real32_T in3_re;
      in3_re = in3->data[i1 * stride_1_0 + in3->size[0] * aux_1_1].re;
      in3_im = -in3->data[i1 * stride_1_0 + in3->size[0] * aux_1_1].im;
      in1->data[i1 + in1->size[0] * i].re =
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].re * in3_re -
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].im * in3_im;
      in1->data[i1 + in1->size[0] * i].im =
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].re * in3_im +
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].im * in3_re;
    }
    aux_1_1 += stride_1_1;
    aux_0_1 += stride_0_1;
  }
}

static real32_T rt_hypotf_snf(real32_T u0, real32_T u1)
{
  real32_T a;
  real32_T b;
  real32_T y;
  a = std::abs(u0);
  b = std::abs(u1);
  if (a < b) {
    a /= b;
    y = b * std::sqrt(a * a + 1.0F);
  } else if (a > b) {
    b /= a;
    y = a * std::sqrt(b * b + 1.0F);
  } else if (std::isnan(b)) {
    y = rtNaNF;
  } else {
    y = a * 1.41421354F;
  }
  return y;
}

static __global__ __launch_bounds__(1024, 1) void wcoherence_kernel26(
    const int32_T i, const int32_T b, emxArray_real32_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    int32_T j2;
    real32_T xtmp;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

//
//
namespace coder {
void wcoherence(const emxArray_real32_T *x, const emxArray_real32_T *y,
                real_T varargin_1, const real_T varargin_3[2],
                real_T varargin_5, emxArray_real32_T *wcoh,
                emxArray_creal32_T *cpu_wcs, boolean_T *wcs_outdatedOnCpu,
                emxArray_creal32_T *gpu_wcs, boolean_T *wcs_outdatedOnGpu)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal32_T gpu_xposdft;
  emxArray_creal32_T *cfspos;
  emxArray_creal32_T *cpu_xposdft;
  emxArray_creal32_T *cwtx;
  emxArray_creal32_T *r;
  emxArray_real32_T b_gpu_y;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_xv;
  emxArray_real32_T gpu_y;
  emxArray_real32_T *b_cpu_y;
  emxArray_real32_T *coitmp;
  emxArray_real32_T *cpu_x;
  emxArray_real32_T *cpu_xv;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *f;
  emxArray_real32_T *g_y;
  emxArray_real32_T *i_y;
  emxArray_real32_T *j_y;
  emxArray_real32_T *k_y;
  emxArray_real32_T *psihat;
  emxArray_real_T gpu_scales;
  emxArray_real_T *cpu_scales;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *f_y;
  const mxArray *h_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  real_T FLimits[2];
  real_T Norig;
  real_T b_y;
  real_T c_y;
  real_T maxperiod;
  real_T minperiod;
  real_T ns;
  int32_T b_i;
  int32_T i;
  int32_T i3;
  int32_T i5;
  int32_T k;
  int32_T nd2;
  char_T unusedExpr[23];
  boolean_T exitg1;
  boolean_T guard1;
  boolean_T scales_outdatedOnGpu;
  boolean_T xposdft_outdatedOnCpu;
  boolean_T xposdft_outdatedOnGpu;
  boolean_T xv_outdatedOnCpu;
  boolean_T xv_outdatedOnGpu;
  gpuEmxReset_real32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&gpu_y);
  gpuEmxReset_real32_T(&gpu_x);
  gpuEmxReset_creal32_T(&gpu_xposdft);
  gpuEmxReset_real32_T(&gpu_xv);
  gpuEmxReset_real_T(&gpu_scales);
  xposdft_outdatedOnCpu = false;
  xposdft_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInitStruct_cwtfilterbank(&fb, &ob_emlrtRTEI, true);
  b_y = std::trunc(b_log2(static_cast<real_T>(x->size[0])));
  FLimits[0] = varargin_3[0];
  FLimits[1] = varargin_3[1];
  c_y = std::trunc(b_log2(static_cast<real_T>(x->size[0])));
  if (!(b_y - 1.0 <= c_y - 1.0)) {
    d_y = nullptr;
    m = emlrtCreateCharArray(2, &iv[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
    emlrtAssign(&d_y, m);
    e_y = nullptr;
    m1 = emlrtCreateDoubleScalar(c_y - 1.0);
    emlrtAssign(&e_y, m1);
    emlrt_marshallIn(b_sprintf(d_y, e_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  guard1 = false;
  if ((varargin_1 == 0.0) || std::isnan(varargin_1)) {
    xposdft_outdatedOnCpu = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k < 2)) {
      if (!(varargin_3[k] == 0.0)) {
        xposdft_outdatedOnCpu = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    if (xposdft_outdatedOnCpu) {
      wavelet::internal::cwt::cwtfreqlimits(static_cast<real_T>(x->size[0]),
                                            10.0, 1.0, varargin_5, &maxperiod,
                                            &Norig, &ns, &c_y, &minperiod);
      FLimits[0] = rt_powd_snf(2.0, -(b_y - 1.0)) * c_y;
      FLimits[1] = c_y;
    } else {
      guard1 = true;
    }
  } else {
    guard1 = true;
  }
  if (guard1 && ((!(varargin_1 == 0.0)) && (!std::isnan(varargin_1)))) {
    xposdft_outdatedOnCpu = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k < 2)) {
      if (!(varargin_3[k] == 0.0)) {
        xposdft_outdatedOnCpu = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    if (xposdft_outdatedOnCpu) {
      wavelet::internal::cwt::cwtfreqlimits(
          static_cast<real_T>(x->size[0]), 10.0, varargin_1, varargin_5,
          &maxperiod, &Norig, &ns, &c_y, &minperiod);
      FLimits[0] = rt_powd_snf(2.0, -(b_y - 1.0)) * c_y;
      FLimits[1] = c_y;
    }
  }
  cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(x->size[0]), FLimits,
                              varargin_1, varargin_5);
  emxInit_real_T(&cpu_scales, 2, &fb_emlrtRTEI, true);
  scales_outdatedOnGpu = false;
  i = cpu_scales->size[0] * cpu_scales->size[1];
  cpu_scales->size[0] = 1;
  cpu_scales->size[1] = fb.Scales->size[1];
  emxEnsureCapacity_real_T(cpu_scales, i, &fb_emlrtRTEI);
  for (i = 0; i < fb.Scales->size[1]; i++) {
    cpu_scales->data[i] = fb.Scales->data[i];
    scales_outdatedOnGpu = true;
  }
  ns = std::fmin(std::floor(static_cast<real_T>(cpu_scales->size[1]) / 2.0),
                 varargin_5);
  emxInit_creal32_T(&cwtx, 2, &pb_emlrtRTEI, true);
  emxInit_real32_T(&f, 1, &pb_emlrtRTEI, true);
  emxInit_real32_T(&coitmp, 1, &pb_emlrtRTEI, true);
  cwtfilterbank_wt(&fb, x, cwtx, f, coitmp);
  emxFree_real32_T(&coitmp);
  emxFree_real32_T(&f);
  emxInit_real32_T(&psihat, 2, &gb_emlrtRTEI, true);
  i = psihat->size[0] * psihat->size[1];
  psihat->size[0] = fb.PsiDFT->size[0];
  psihat->size[1] = fb.PsiDFT->size[1];
  emxEnsureCapacity_real32_T(psihat, i, &gb_emlrtRTEI);
  for (i = 0; i < fb.PsiDFT->size[0] * fb.PsiDFT->size[1]; i++) {
    psihat->data[i] = static_cast<real32_T>(fb.PsiDFT->data[i]);
  }
  emxInit_real32_T(&cpu_xv, 2, &hb_emlrtRTEI, true);
  xv_outdatedOnCpu = false;
  xv_outdatedOnGpu = false;
  i = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = y->size[0];
  emxEnsureCapacity_real32_T(cpu_xv, i, &hb_emlrtRTEI);
  for (i = 0; i < y->size[0]; i++) {
    cpu_xv->data[i] = y->data[i];
    xv_outdatedOnGpu = true;
  }
  Norig = fb.SignalLength;
  if (fb.SignalPad > 0.0) {
    boolean_T validLaunchParams;
    c_y = fb.SignalPad;
    if (c_y < 1.0) {
      b_i = 0;
    } else {
      b_i = static_cast<int32_T>(c_y);
    }
    c_y = (static_cast<real_T>(y->size[0]) - fb.SignalPad) + 1.0;
    if (c_y > y->size[0]) {
      k = 0;
      i3 = 1;
      i5 = -1;
    } else {
      k = y->size[0] - 1;
      i3 = -1;
      i5 = static_cast<int32_T>(c_y) - 1;
    }
    emxInit_real32_T(&cpu_x, 2, &ib_emlrtRTEI, true);
    xposdft_outdatedOnCpu = false;
    i = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = 1;
    cpu_x->size[1] = b_i;
    emxEnsureCapacity_real32_T(cpu_x, i, &ib_emlrtRTEI);
    for (i = 0; i < b_i; i++) {
      cpu_x->data[i] = y->data[i];
      xposdft_outdatedOnCpu = true;
    }
    mwGetLaunchParameters1D(computeNumIters((b_i >> 1) - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !xposdft_outdatedOnCpu);
    if (xposdft_outdatedOnCpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      wcoherence_kernel26<<<grid, block>>>(b_i, (b_i >> 1) - 1, gpu_x);
    }
    xposdft_outdatedOnCpu = true;
    xv_outdatedOnGpu = false;
    i = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    cpu_xv->size[1] = ((y->size[0] + cpu_x->size[1]) + div_s32(i5 - k, i3)) + 1;
    emxEnsureCapacity_real32_T(cpu_xv, i, &lb_emlrtRTEI);
    b_i = cpu_x->size[1];
    for (i = 0; i < b_i; i++) {
      if (xposdft_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_x, &gpu_x);
      }
      xposdft_outdatedOnCpu = false;
      cpu_xv->data[i] = cpu_x->data[i];
      xv_outdatedOnGpu = true;
    }
    b_i = y->size[0];
    for (i = 0; i < b_i; i++) {
      cpu_xv->data[i + cpu_x->size[1]] = y->data[i];
      xv_outdatedOnGpu = true;
    }
    b_i = div_s32(i5 - k, i3);
    for (i = 0; i <= b_i; i++) {
      cpu_xv->data[(i + y->size[0]) + cpu_x->size[1]] = y->data[k + i3 * i];
      xv_outdatedOnGpu = true;
    }
    emxFree_real32_T(&cpu_x);
  }
  emxInit_creal32_T(&cpu_xposdft, 2, &qb_emlrtRTEI, true);
  fft(cpu_xv, &xv_outdatedOnCpu, &gpu_xv, &xv_outdatedOnGpu, cpu_xposdft,
      &xposdft_outdatedOnCpu, &gpu_xposdft, &xposdft_outdatedOnGpu);
  emxFree_real32_T(&cpu_xv);
  emxInit_creal32_T(&r, 2, &vb_emlrtRTEI, true);
  if (xposdft_outdatedOnCpu) {
    gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xposdft, &gpu_xposdft);
  }
  bsxfun(cpu_xposdft, psihat, r);
  emxFree_creal32_T(&cpu_xposdft);
  emxFree_real32_T(&psihat);
  emxInit_creal32_T(&cfspos, 2, &rb_emlrtRTEI, true);
  ifft(r, cfspos);
  emxFree_creal32_T(&r);
  if (fb.SignalPad > 0.0) {
    maxperiod = fb.SignalPad + 1.0;
    c_y = fb.SignalPad + Norig;
    if (maxperiod > c_y) {
      nd2 = 0;
      k = 0;
    } else {
      nd2 = static_cast<int32_T>(maxperiod) - 1;
      k = static_cast<int32_T>(c_y);
    }
    i3 = cfspos->size[0];
    b_i = k - nd2;
    for (i = 0; i < b_i; i++) {
      for (i5 = 0; i5 < i3; i5++) {
        cfspos->data[i5 + i3 * i] =
            cfspos->data[i5 + cfspos->size[0] * (nd2 + i)];
      }
    }
    i = cfspos->size[0] * cfspos->size[1];
    cfspos->size[1] = k - nd2;
    emxEnsureCapacity_creal32_T(cfspos, i, &jb_emlrtRTEI);
  }
  nd2 = static_cast<int32_T>(
      std::trunc(static_cast<real_T>(cwtx->size[0]) / 2.0));
  if (!(ns <= nd2)) {
    f_y = nullptr;
    m2 = emlrtCreateCharArray(2, &iv1[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
    emlrtAssign(&f_y, m2);
    h_y = nullptr;
    m3 = emlrtCreateDoubleScalar(static_cast<real_T>(nd2));
    emlrtAssign(&h_y, m3);
    emlrt_marshallIn(b_sprintf(f_y, h_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  nd2 = cwtx->size[0] * cwtx->size[1];
  emxInit_real32_T(&g_y, 2, &sb_emlrtRTEI, true);
  i = g_y->size[0] * g_y->size[1];
  g_y->size[0] = cwtx->size[0];
  g_y->size[1] = cwtx->size[1];
  emxEnsureCapacity_real32_T(g_y, i, &h_emlrtRTEI);
  for (k = 0; k < nd2; k++) {
    g_y->data[k] = rt_hypotf_snf(cwtx->data[k].re, cwtx->data[k].im);
  }
  emxInit_real32_T(&cpu_y, 2, &sb_emlrtRTEI, true);
  xposdft_outdatedOnGpu = false;
  xposdft_outdatedOnCpu = false;
  i = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = g_y->size[0];
  cpu_y->size[1] = g_y->size[1];
  emxEnsureCapacity_real32_T(cpu_y, i, &kb_emlrtRTEI);
  nd2 = g_y->size[0] * g_y->size[1];
  for (k = 0; k < nd2; k++) {
    cpu_y->data[k] = g_y->data[k] * g_y->data[k];
    xposdft_outdatedOnCpu = true;
  }
  emxFree_real32_T(&g_y);
  wavelet::internal::cwt::smoothCFS(cpu_y, &xposdft_outdatedOnGpu, &gpu_y,
                                    &xposdft_outdatedOnCpu, cpu_scales,
                                    &gpu_scales, &scales_outdatedOnGpu, ns);
  nd2 = cfspos->size[0] * cfspos->size[1];
  emxInit_real32_T(&i_y, 2, &tb_emlrtRTEI, true);
  i = i_y->size[0] * i_y->size[1];
  i_y->size[0] = cfspos->size[0];
  i_y->size[1] = cfspos->size[1];
  emxEnsureCapacity_real32_T(i_y, i, &h_emlrtRTEI);
  for (k = 0; k < nd2; k++) {
    i_y->data[k] = rt_hypotf_snf(cfspos->data[k].re, cfspos->data[k].im);
  }
  emxInit_real32_T(&b_cpu_y, 2, &tb_emlrtRTEI, true);
  xv_outdatedOnGpu = false;
  xposdft_outdatedOnCpu = false;
  i = b_cpu_y->size[0] * b_cpu_y->size[1];
  b_cpu_y->size[0] = i_y->size[0];
  b_cpu_y->size[1] = i_y->size[1];
  emxEnsureCapacity_real32_T(b_cpu_y, i, &kb_emlrtRTEI);
  nd2 = i_y->size[0] * i_y->size[1];
  for (k = 0; k < nd2; k++) {
    b_cpu_y->data[k] = i_y->data[k] * i_y->data[k];
    xposdft_outdatedOnCpu = true;
  }
  emxFree_real32_T(&i_y);
  wavelet::internal::cwt::smoothCFS(b_cpu_y, &xv_outdatedOnGpu, &b_gpu_y,
                                    &xposdft_outdatedOnCpu, cpu_scales,
                                    &gpu_scales, &scales_outdatedOnGpu, ns);
  if ((cwtx->size[0] == cfspos->size[0]) &&
      (cwtx->size[1] == cfspos->size[1])) {
    *wcs_outdatedOnCpu = false;
    *wcs_outdatedOnGpu = false;
    i = cpu_wcs->size[0] * cpu_wcs->size[1];
    cpu_wcs->size[0] = cwtx->size[0];
    cpu_wcs->size[1] = cwtx->size[1];
    emxEnsureCapacity_creal32_T(cpu_wcs, i, &mb_emlrtRTEI);
    for (i = 0; i < cwtx->size[0] * cwtx->size[1]; i++) {
      real32_T cfspos_im;
      real32_T cfspos_re;
      cfspos_re = cfspos->data[i].re;
      cfspos_im = -cfspos->data[i].im;
      cpu_wcs->data[i].re =
          cwtx->data[i].re * cfspos_re - cwtx->data[i].im * cfspos_im;
      cpu_wcs->data[i].im =
          cwtx->data[i].re * cfspos_im + cwtx->data[i].im * cfspos_re;
      *wcs_outdatedOnGpu = true;
    }
  } else {
    if (*wcs_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_creal32_T(cpu_wcs, gpu_wcs);
    }
    binary_expand_op_1(cpu_wcs, cwtx, cfspos);
    *wcs_outdatedOnCpu = false;
    *wcs_outdatedOnGpu = true;
  }
  emxFree_creal32_T(&cfspos);
  emxFree_creal32_T(&cwtx);
  wavelet::internal::cwt::b_smoothCFS(cpu_wcs, wcs_outdatedOnCpu, gpu_wcs,
                                      wcs_outdatedOnGpu, cpu_scales,
                                      &gpu_scales, &scales_outdatedOnGpu, ns);
  emxFree_real_T(&cpu_scales);
  nd2 = cpu_wcs->size[0] * cpu_wcs->size[1];
  emxInit_real32_T(&j_y, 2, &ub_emlrtRTEI, true);
  i = j_y->size[0] * j_y->size[1];
  j_y->size[0] = cpu_wcs->size[0];
  j_y->size[1] = cpu_wcs->size[1];
  emxEnsureCapacity_real32_T(j_y, i, &h_emlrtRTEI);
  for (k = 0; k < nd2; k++) {
    if (*wcs_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_creal32_T(cpu_wcs, gpu_wcs);
    }
    *wcs_outdatedOnCpu = false;
    j_y->data[k] = rt_hypotf_snf(cpu_wcs->data[k].re, cpu_wcs->data[k].im);
  }
  emxInit_real32_T(&k_y, 2, &ub_emlrtRTEI, true);
  i = k_y->size[0] * k_y->size[1];
  k_y->size[0] = j_y->size[0];
  k_y->size[1] = j_y->size[1];
  emxEnsureCapacity_real32_T(k_y, i, &kb_emlrtRTEI);
  nd2 = j_y->size[0] * j_y->size[1];
  for (k = 0; k < nd2; k++) {
    k_y->data[k] = j_y->data[k] * j_y->data[k];
  }
  emxFree_real32_T(&j_y);
  if (cpu_y->size[0] == 1) {
    i3 = b_cpu_y->size[0];
  } else {
    i3 = cpu_y->size[0];
  }
  if (cpu_y->size[1] == 1) {
    nd2 = b_cpu_y->size[1];
  } else {
    nd2 = cpu_y->size[1];
  }
  if ((cpu_y->size[0] == b_cpu_y->size[0]) &&
      (cpu_y->size[1] == b_cpu_y->size[1]) && (k_y->size[0] == i3) &&
      (k_y->size[1] == nd2)) {
    i = wcoh->size[0] * wcoh->size[1];
    wcoh->size[0] = k_y->size[0];
    wcoh->size[1] = k_y->size[1];
    emxEnsureCapacity_real32_T(wcoh, i, &nb_emlrtRTEI);
    for (i = 0; i < k_y->size[0] * k_y->size[1]; i++) {
      if (xposdft_outdatedOnGpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &gpu_y);
      }
      xposdft_outdatedOnGpu = false;
      if (xv_outdatedOnGpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_y, &b_gpu_y);
      }
      xv_outdatedOnGpu = false;
      wcoh->data[i] = k_y->data[i] / (cpu_y->data[i] * b_cpu_y->data[i]);
    }
  } else {
    if (xposdft_outdatedOnGpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &gpu_y);
    }
    if (xv_outdatedOnGpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_y, &b_gpu_y);
    }
    binary_expand_op(wcoh, k_y, cpu_y, b_cpu_y);
  }
  emxFree_real32_T(&k_y);
  emxFree_real32_T(&b_cpu_y);
  emxFree_real32_T(&cpu_y);
  nd2 = wcoh->size[0] * wcoh->size[1] - 1;
  for (b_i = 0; b_i <= nd2; b_i++) {
    if (wcoh->data[b_i] > 1.0F) {
      wcoh->data[b_i] = 1.0F;
    }
  }
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real_T(&gpu_scales);
  gpuEmxFree_real32_T(&gpu_xv);
  gpuEmxFree_creal32_T(&gpu_xposdft);
  gpuEmxFree_real32_T(&gpu_x);
  gpuEmxFree_real32_T(&gpu_y);
  gpuEmxFree_real32_T(&b_gpu_y);
}

} // namespace coder

// End of code generation (wcoherence.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// conv2.cu
//
// Code generation for function 'conv2'
//

// Include files
#include "conv2.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "rt_nonfinite.h"
#include "MWCUBLASUtils.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWErrorCodeUtils.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo wf_emlrtRTEI{
    56,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo xf_emlrtRTEI{
    47,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo ag_emlrtRTEI{
    49,                                                               // lineNo
    9,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

static emlrtRTEInfo
    fg_emlrtRTEI{
        158,      // lineNo
        24,       // colNo
        "mtimes", // fName
        "/usr/local/MATLAB/R2024a/toolbox/eml/eml/+coder/+internal/+blas/"
        "mtimes.m" // pName
    };

static emlrtRTEInfo gg_emlrtRTEI{
    1,                  // lineNo
    1,                  // colNo
    "gpu_conv2_kernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/gpucoder/gpucoder/+coder/+internal/"
    "+images/gpu_conv2_kernel.p" // pName
};

static emlrtRTEInfo jg_emlrtRTEI{
    58,                                                               // lineNo
    5,                                                                // colNo
    "conv2",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/conv2.m" // pName
};

// Function Declarations
namespace coder {
static void
b_conv2NonSeparable(emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
                    emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
                    emxArray_real32_T *cpu_b, boolean_T *b_outdatedOnCpu,
                    emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
                    emxArray_real32_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_real32_T *gpu_c, boolean_T *c_outdatedOnGpu);

static void
c_conv2NonSeparable(emxArray_creal32_T *cpu_a, emxArray_creal32_T *gpu_a,
                    boolean_T *a_outdatedOnGpu, const emxArray_real32_T *b,
                    emxArray_creal32_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_creal32_T *gpu_c, boolean_T *c_outdatedOnGpu);

static void
conv2NonSeparable(emxArray_real32_T *cpu_a, emxArray_real32_T *gpu_a,
                  boolean_T *a_outdatedOnGpu, emxArray_real32_T *cpu_b,
                  emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
                  emxArray_real32_T *cpu_c, boolean_T *c_outdatedOnCpu,
                  emxArray_real32_T *gpu_c, boolean_T *c_outdatedOnGpu);

} // namespace coder
static __global__ void conv2NonSeparable_kernel121(const int32_T b,
                                                   emxArray_creal32_T c);

static __global__ void conv2NonSeparable_kernel122(const int32_T b,
                                                   emxArray_creal32_T c);

static __global__ void conv2NonSeparable_kernel123(const real32_T cv_re,
                                                   creal32_T b_data[1]);

static __global__ void conv2NonSeparable_kernel124(const emxArray_creal32_T a,
                                                   const creal32_T b_data[1],
                                                   emxArray_creal32_T b_a);

static __global__ void conv2NonSeparable_kernel125(const emxArray_creal32_T a,
                                                   const int32_T iv,
                                                   emxArray_creal32_T c);

static __global__ void conv2NonSeparable_kernel126(const int32_T OH,
                                                   emxArray_creal32_T expanded);

static __global__ void conv2NonSeparable_kernel127(const int32_T offsetH,
                                                   const emxArray_int32_T y,
                                                   const int32_T b_y,
                                                   emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel128(const emxArray_int32_T y,
                                                   const int32_T b_y,
                                                   emxArray_int32_T iv1);

static __global__ void conv2NonSeparable_kernel129(
    const emxArray_creal32_T a, const int32_T iv_dim0,
    const emxArray_int32_T iv1, const emxArray_int32_T iv, const int32_T b_iv,
    const int32_T c_iv, emxArray_creal32_T expanded, int32_T expanded_dim0);

static __global__ void
conv2NonSeparable_kernel130(const int32_T offsetH,
                            const emxArray_creal32_T expanded, const int32_T k,
                            const emxArray_int32_T rows, const int32_T b_rows,
                            emxArray_creal32_T newIm, int32_T expanded_dim0);

static __global__ void conv2NonSeparable_kernel131(const real32_T cv_re,
                                                   const int32_T offsetH,
                                                   const int32_T k,
                                                   emxArray_creal32_T c,
                                                   int32_T c_dim0);

static __global__ void conv2NonSeparable_kernel132(const real32_T cv_im,
                                                   const int32_T offsetH,
                                                   const int32_T k,
                                                   emxArray_creal32_T c,
                                                   int32_T c_dim0);

static __global__ void conv2NonSeparable_kernel59(const int32_T b,
                                                  emxArray_real32_T c);

static __global__ void conv2NonSeparable_kernel60(const int32_T b,
                                                  emxArray_real32_T c);

static __global__ void conv2NonSeparable_kernel61(const emxArray_real32_T c,
                                                  const int32_T b,
                                                  emxArray_real32_T b_c);

static __global__ void conv2NonSeparable_kernel62(const int32_T OH,
                                                  emxArray_real32_T expanded);

static __global__ void conv2NonSeparable_kernel63(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel64(const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv1);

static __global__ void conv2NonSeparable_kernel65(
    const emxArray_real32_T a, const int32_T iv_dim0,
    const emxArray_int32_T iv1, const emxArray_int32_T iv, const int32_T b_iv,
    const int32_T c_iv, emxArray_real32_T expanded, int32_T expanded_dim0);

static __global__ void conv2NonSeparable_kernel66(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_real32_T b, const int32_T c, const int32_T d,
    emxArray_real32_T b_c, int32_T b_dim0, int32_T expanded_dim0,
    int32_T c_dim0);

static __global__ void conv2NonSeparable_kernel67(const int32_T b,
                                                  emxArray_real32_T c);

static __global__ void conv2NonSeparable_kernel68(const int32_T OH,
                                                  emxArray_real32_T expanded);

static __global__ void conv2NonSeparable_kernel69(const int32_T offsetH,
                                                  const emxArray_int32_T y,
                                                  const int32_T b_y,
                                                  emxArray_int32_T iv);

static __global__ void conv2NonSeparable_kernel70(const emxArray_real32_T a,
                                                  const emxArray_int32_T iv,
                                                  const int32_T b,
                                                  emxArray_real32_T expanded);

static __global__ void
conv2NonSeparable_kernel71(const emxArray_real32_T expanded,
                           const emxArray_int32_T rows,
                           const emxArray_real32_T b, const int32_T c,
                           emxArray_real32_T b_c, int32_T b_dim0);

static __global__ void conv2_kernel104(const emxArray_creal32_T a,
                                       const int32_T b_a,
                                       emxArray_creal32_T c_a, int32_T a_dim0);

static __global__ void conv2_kernel105(const int32_T a, emxArray_creal32_T c);

static __global__ void conv2_kernel106(const emxArray_creal32_T a,
                                       const real32_T b, real32_T *f1,
                                       real32_T *f);

static __global__ void conv2_kernel107(const real32_T *f, const real32_T *f1,
                                       emxArray_creal32_T c);

static __global__ void conv2_kernel108(const emxArray_creal32_T c,
                                       const int32_T b_c,
                                       emxArray_creal32_T c_c, int32_T c_dim0);

static __global__ void conv2_kernel109(const int32_T a,
                                       emxArray_creal32_T expanded);

static __global__ void conv2_kernel110(const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel111(const emxArray_creal32_T a,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_creal32_T expanded);

static __global__ void conv2_kernel112(const emxArray_creal32_T expanded,
                                       const real32_T b, const int32_T c,
                                       emxArray_creal32_T b_c);

static __global__ void conv2_kernel113(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel114(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel115(const real32_T f, creal32_T b_data[1]);

static __global__ void conv2_kernel116(const emxArray_creal32_T a,
                                       const creal32_T b_data[1],
                                       emxArray_creal32_T c);

static __global__ void conv2_kernel117(const int32_T OH,
                                       emxArray_real32_T expanded);

static __global__ void conv2_kernel118(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv1);

static __global__ void
conv2_kernel119(const emxArray_real32_T b, const int32_T mc,
                const emxArray_int32_T iv1, const int32_T c,
                emxArray_real32_T expanded, int32_T expanded_dim0);

static __global__ void
conv2_kernel120(const emxArray_real32_T expanded, const emxArray_int32_T cols,
                const emxArray_int32_T rows, const emxArray_creal32_T a,
                const int32_T b, const int32_T c, emxArray_creal32_T b_c,
                int32_T a_dim0, int32_T a_dim1, int32_T expanded_dim0,
                int32_T c_dim0);

static __global__ void conv2_kernel133(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel134(const int32_T OH,
                                       emxArray_real32_T expanded);

static __global__ void conv2_kernel135(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv1);

static __global__ void conv2_kernel136(const emxArray_real32_T b,
                                       const emxArray_int32_T iv1,
                                       const int32_T c,
                                       emxArray_real32_T expanded);

static __global__ void conv2_kernel137(const emxArray_real32_T expanded,
                                       const emxArray_int32_T rows,
                                       const emxArray_creal32_T a,
                                       const int32_T b, emxArray_creal32_T c,
                                       int32_T a_dim0);

static __global__ void conv2_kernel138(const int32_T b, emxArray_creal32_T c);

static __global__ void conv2_kernel139(const int32_T OH,
                                       emxArray_creal32_T expanded);

static __global__ void conv2_kernel140(const int32_T offsetH,
                                       const emxArray_int32_T y,
                                       const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel141(const emxArray_creal32_T a,
                                       const emxArray_int32_T iv,
                                       const int32_T b,
                                       emxArray_creal32_T expanded);

static __global__ void conv2_kernel142(const emxArray_creal32_T expanded,
                                       const int32_T offsetH,
                                       const emxArray_int32_T rows,
                                       const int32_T b_rows,
                                       emxArray_creal32_T newIm);

static __global__ void conv2_kernel143(const real32_T cv_im,
                                       const real32_T cv_re,
                                       const int32_T offsetH,
                                       emxArray_creal32_T c);

static __global__ void conv2_kernel44(const emxArray_real32_T a,
                                      const int32_T b_a, emxArray_real32_T c_a,
                                      int32_T a_dim0);

static __global__ void conv2_kernel45(const int32_T a, emxArray_real32_T c);

static __global__ void conv2_kernel46(const real32_T b,
                                      const emxArray_real32_T a,
                                      emxArray_real32_T c);

static __global__ void conv2_kernel47(const emxArray_real32_T c,
                                      const int32_T b_c, emxArray_real32_T c_c,
                                      int32_T c_dim0);

static __global__ void conv2_kernel48(const int32_T a,
                                      emxArray_real32_T expanded);

static __global__ void conv2_kernel49(const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv);

static __global__ void conv2_kernel50(const emxArray_real32_T a,
                                      const emxArray_int32_T iv,
                                      const int32_T b,
                                      emxArray_real32_T expanded);

static __global__ void conv2_kernel51(const real32_T b,
                                      const emxArray_real32_T expanded,
                                      const int32_T c, emxArray_real32_T b_c);

static __global__ void conv2_kernel52(const int32_T b, emxArray_real32_T c);

static __global__ void conv2_kernel53(const int32_T b, emxArray_real32_T c);

static __global__ void conv2_kernel54(const emxArray_real32_T a,
                                      const emxArray_real32_T b,
                                      emxArray_real32_T c);

static __global__ void conv2_kernel55(const int32_T OH,
                                      emxArray_real32_T expanded);

static __global__ void conv2_kernel56(const int32_T offsetH,
                                      const emxArray_int32_T y,
                                      const int32_T b_y, emxArray_int32_T iv1);

static __global__ void
conv2_kernel57(const emxArray_real32_T b, const int32_T mc,
               const emxArray_int32_T iv1, const int32_T c,
               emxArray_real32_T expanded, int32_T expanded_dim0);

static __global__ void
conv2_kernel58(const emxArray_real32_T expanded, const emxArray_int32_T cols,
               const emxArray_int32_T rows, const emxArray_real32_T a,
               const int32_T b, const int32_T c, emxArray_real32_T b_c,
               int32_T a_dim0, int32_T a_dim1, int32_T expanded_dim0,
               int32_T c_dim0);

static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line);

// Function Definitions
//
//
namespace coder {
static void
b_conv2NonSeparable(emxArray_real32_T *cpu_a, boolean_T *a_outdatedOnCpu,
                    emxArray_real32_T *gpu_a, boolean_T *a_outdatedOnGpu,
                    emxArray_real32_T *cpu_b, boolean_T *b_outdatedOnCpu,
                    emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
                    emxArray_real32_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_real32_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *cpu_expanded;
  int32_T k;
  int32_T mc;
  boolean_T validLaunchParams;
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_real32_T(&gpu_expanded);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  k = cpu_c->size[0];
  cpu_c->size[0] = mc;
  emxEnsureCapacity_real32_T(cpu_c, k, &yf_emlrtRTEI);
  mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    conv2NonSeparable_kernel67<<<grid, block>>>(mc - 1, *gpu_c);
  }
  *c_outdatedOnGpu = false;
  *c_outdatedOnCpu = true;
  emxInit_real32_T(&cpu_expanded, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &bg_emlrtRTEI, true);
  if ((cpu_a->size[0] != 0) && (cpu_b->size[0] != 0)) {
    if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
      real32_T b;
      if (*b_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      b = cpu_b->data[0];
      k = cpu_c->size[0];
      cpu_c->size[0] = 1;
      emxEnsureCapacity_real32_T(cpu_c, k, &yf_emlrtRTEI);
      if (*a_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_a, gpu_a);
      }
      *a_outdatedOnCpu = false;
      cpu_c->data[0] = cpu_a->data[0] * b;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      real_T blockDims_idx_0;
      int32_T b_OH;
      int32_T n;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims_idx_0;
      OH = (static_cast<uint32_T>(cpu_a->size[0]) +
            static_cast<uint32_T>(cpu_b->size[0])) -
           1U;
      mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      blockDims_idx_0 = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
          static_cast<real_T>(mc));
      threadDims_idx_0 = static_cast<int8_T>(mc);
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
      if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_b->size[0];
      }
      k = cpu_expanded->size[0];
      cpu_expanded->size[0] = mc - 1;
      emxEnsureCapacity_real32_T(cpu_expanded, k, &bg_emlrtRTEI);
      mc = (static_cast<int32_T>(OH) + cpu_b->size[0]) - 2;
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel68<<<grid, block>>>(mc, gpu_expanded);
      }
      n = cpu_a->size[0];
      k = cpu_y->size[0] * cpu_y->size[1];
      cpu_y->size[0] = 1;
      cpu_y->size[1] = cpu_a->size[0];
      emxEnsureCapacity_int32_T(cpu_y, k, &cg_emlrtRTEI);
      cpu_y->data[0] = 1;
      mc = 1;
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_y->data[k + 1] = mc;
      }
      k = cpu_iv->size[0];
      cpu_iv->size[0] = cpu_y->size[1];
      emxEnsureCapacity_int32_T(cpu_iv, k, &bg_emlrtRTEI);
      mc = cpu_y->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(mc), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
      gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel69<<<grid, block>>>(offsetH, gpu_y, mc, gpu_iv);
      }
      mc = cpu_iv->size[0];
      mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
      if (*a_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
      }
      *a_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel70<<<grid, block>>>(*gpu_a, gpu_iv, mc - 1,
                                                    gpu_expanded);
      }
      n = cpu_b->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_b->size[0];
      emxEnsureCapacity_int32_T(cpu_rows, k, &cg_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      for (k = 0; k <= n - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      k = cpu_c->size[0];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      emxEnsureCapacity_real32_T(cpu_c, k, &bg_emlrtRTEI);
      if (blockDims_idx_0 < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims_idx_0);
      } else {
        OH = MAX_uint32_T;
      }
      mwApplyLaunchParameters(
          computeNumIters(b_OH - 1), dim3(OH, 1U, 1U),
          dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid, &block);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2NonSeparable_kernel71<<<grid, block>>>(
            gpu_expanded, gpu_rows, *gpu_b, b_OH - 1, *gpu_c, cpu_b->size[0U]);
      }
    }
  }
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&cpu_y);
  emxFree_real32_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_rows);
}

//
//
static void
c_conv2NonSeparable(emxArray_creal32_T *cpu_a, emxArray_creal32_T *gpu_a,
                    boolean_T *a_outdatedOnGpu, const emxArray_real32_T *b,
                    emxArray_creal32_T *cpu_c, boolean_T *c_outdatedOnCpu,
                    emxArray_creal32_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal32_T b_gpu_a;
  emxArray_creal32_T gpu_expanded;
  emxArray_creal32_T gpu_newIm;
  emxArray_creal32_T *b_cpu_a;
  emxArray_creal32_T *cpu_expanded;
  emxArray_creal32_T *cpu_newIm;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  creal32_T(*gpu_b_data)[1];
  int32_T i;
  int32_T mc;
  int32_T nc;
  boolean_T validLaunchParams;
  checkCudaError(mwCudaMalloc(&gpu_b_data, 8UL), __FILE__, __LINE__);
  gpuEmxReset_creal32_T(&b_gpu_a);
  gpuEmxReset_creal32_T(&gpu_newIm);
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_creal32_T(&gpu_expanded);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (b->size[0] == 0)) {
    mc = cpu_a->size[0] + b->size[0];
  } else {
    mc = (cpu_a->size[0] + b->size[0]) - 1;
  }
  if (cpu_a->size[1] == 0) {
    nc = 1;
  } else {
    nc = cpu_a->size[1];
  }
  i = cpu_c->size[0] * cpu_c->size[1];
  cpu_c->size[0] = mc;
  cpu_c->size[1] = nc;
  emxEnsureCapacity_creal32_T(cpu_c, i, &yf_emlrtRTEI);
  i = mc * nc - 1;
  mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    conv2NonSeparable_kernel121<<<grid, block>>>(i, *gpu_c);
  }
  emxInit_creal32_T(&cpu_expanded, 2, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &bg_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_newIm, 1, &bg_emlrtRTEI, true);
  emxInit_creal32_T(&b_cpu_a, 1, &ig_emlrtRTEI, true);
  if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0) || (b->size[0] == 0)) {
    i = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_creal32_T(cpu_c, i, &yf_emlrtRTEI);
    i = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel122<<<grid, block>>>(i, *gpu_c);
    }
  } else if ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1) &&
             (b->size[0] == 1)) {
    real32_T cv_re;
    cv_re = b->data[0];
    conv2NonSeparable_kernel123<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        cv_re, *gpu_b_data);
    i = b_cpu_a->size[0];
    b_cpu_a->size[0] = 1;
    emxEnsureCapacity_creal32_T(b_cpu_a, i, &ig_emlrtRTEI);
    gpuEmxEnsureCapacity_creal32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    gpuEmxEnsureCapacity_creal32_T(b_cpu_a, &b_gpu_a, true);
    if (*a_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    conv2NonSeparable_kernel124<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        *gpu_a, *gpu_b_data, b_gpu_a);
    i = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_creal32_T(cpu_c, i, &yf_emlrtRTEI);
    i = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel125<<<grid, block>>>(b_gpu_a, i, *gpu_c);
    }
  } else {
    int32_T iv[2];
    int32_T OW;
    int32_T b_OH;
    int32_T offsetH;
    uint32_T OH;
    boolean_T c_needsGpuEnsureCapacity;
    boolean_T rows_needsGpuEnsureCapacity;
    boolean_T rows_outdatedOnGpu;
    OH = (static_cast<uint32_T>(cpu_a->size[0]) +
          static_cast<uint32_T>(b->size[0])) -
         1U;
    OW = cpu_a->size[1];
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    b_OH = static_cast<int32_T>(OH);
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(b->size[0]) / 2.0)) +
              static_cast<int32_T>(
                  std::floor((static_cast<real_T>(b->size[0]) - 1.0) / 2.0));
    if (b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
      mc = MAX_int32_T;
    } else {
      mc = static_cast<int32_T>(OH) + b->size[0];
    }
    i = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[0] = mc - 1;
    emxEnsureCapacity_creal32_T(cpu_expanded, i, &bg_emlrtRTEI);
    if (cpu_a->size[1] > 2147483646) {
      mc = MAX_int32_T;
    } else {
      mc = cpu_a->size[1] + 1;
    }
    i = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[1] = mc - 1;
    emxEnsureCapacity_creal32_T(cpu_expanded, i, &bg_emlrtRTEI);
    i = ((static_cast<int32_T>(OH) + b->size[0]) - 1) * cpu_a->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel126<<<grid, block>>>(i, gpu_expanded);
    }
    nc = cpu_a->size[0];
    i = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_a->size[0];
    emxEnsureCapacity_int32_T(cpu_y, i, &cg_emlrtRTEI);
    cpu_y->data[0] = 1;
    mc = 1;
    for (int32_T k{0}; k <= nc - 2; k++) {
      mc++;
      cpu_y->data[k + 1] = mc;
    }
    i = cpu_iv->size[0];
    cpu_iv->size[0] = cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv, i, &bg_emlrtRTEI);
    i = cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel127<<<grid, block>>>(offsetH, gpu_y, i, gpu_iv);
    }
    nc = cpu_a->size[1];
    i = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_a->size[1];
    emxEnsureCapacity_int32_T(b_cpu_y, i, &cg_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    mc = 1;
    for (int32_T k{0}; k <= nc - 2; k++) {
      mc++;
      b_cpu_y->data[k + 1] = mc;
    }
    i = cpu_iv1->size[0];
    cpu_iv1->size[0] = b_cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv1, i, &bg_emlrtRTEI);
    i = b_cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel128<<<grid, block>>>(b_gpu_y, i, gpu_iv1);
    }
    iv[0] = cpu_iv->size[0];
    iv[1] = cpu_iv1->size[0];
    mwGetLaunchParameters1D(computeNumIters(iv[1] - 1, iv[0] - 1), &grid,
                            &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    if (*a_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel129<<<grid, block>>>(
          *gpu_a, iv[0], gpu_iv1, gpu_iv, iv[0] - 1, iv[1] - 1, gpu_expanded,
          cpu_expanded->size[0U]);
    }
    nc = b->size[0];
    i = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = b->size[0];
    emxEnsureCapacity_int32_T(cpu_rows, i, &cg_emlrtRTEI);
    cpu_rows->data[0] = 0;
    rows_outdatedOnGpu = true;
    rows_needsGpuEnsureCapacity = true;
    mc = 0;
    for (int32_T k{0}; k <= nc - 2; k++) {
      mc++;
      cpu_rows->data[k + 1] = mc;
    }
    i = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = static_cast<int32_T>(OH);
    cpu_c->size[1] = cpu_a->size[1];
    emxEnsureCapacity_creal32_T(cpu_c, i, &bg_emlrtRTEI);
    c_needsGpuEnsureCapacity = true;
    mc = b->size[0];
    for (offsetH = 0; offsetH < OW; offsetH++) {
      for (int32_T k{0}; k < b_OH; k++) {
        real32_T cv_im;
        real32_T cv_re;
        boolean_T newIm_outdatedOnCpu;
        i = cpu_newIm->size[0];
        cpu_newIm->size[0] = cpu_rows->size[1];
        emxEnsureCapacity_creal32_T(cpu_newIm, i, &bg_emlrtRTEI);
        i = cpu_rows->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(i), &grid, &block, 2147483647U);
        if (rows_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows,
                                       !rows_outdatedOnGpu);
        }
        rows_needsGpuEnsureCapacity = false;
        gpuEmxEnsureCapacity_creal32_T(cpu_newIm, &gpu_newIm, true);
        if (rows_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
        }
        rows_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2NonSeparable_kernel130<<<grid, block>>>(offsetH, gpu_expanded, k,
                                                       gpu_rows, i, gpu_newIm,
                                                       cpu_expanded->size[0U]);
        }
        newIm_outdatedOnCpu = true;
        cv_re = 0.0F;
        cv_im = 0.0F;
        for (nc = 0; nc < mc; nc++) {
          if (newIm_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_creal32_T(cpu_newIm, &gpu_newIm);
          }
          cv_re += cpu_newIm->data[nc].re * b->data[(b->size[0] - nc) - 1];
          newIm_outdatedOnCpu = false;
          cv_im += cpu_newIm->data[nc].im * b->data[(b->size[0] - nc) - 1];
        }
        if (c_needsGpuEnsureCapacity) {
          gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
        }
        conv2NonSeparable_kernel131<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            cv_re, offsetH, k, *gpu_c, cpu_c->size[0U]);
        c_needsGpuEnsureCapacity = false;
        conv2NonSeparable_kernel132<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            cv_im, offsetH, k, *gpu_c, cpu_c->size[0U]);
      }
    }
  }
  emxFree_creal32_T(&b_cpu_a);
  emxFree_creal32_T(&cpu_newIm);
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv1);
  emxFree_int32_T(&b_cpu_y);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&cpu_y);
  emxFree_creal32_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_creal32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_int32_T(&gpu_rows);
  gpuEmxFree_creal32_T(&gpu_newIm);
  gpuEmxFree_creal32_T(&b_gpu_a);
  checkCudaError(mwCudaFree(*gpu_b_data), __FILE__, __LINE__);
  *c_outdatedOnCpu = true;
  *c_outdatedOnGpu = false;
}

//
//
static void
conv2NonSeparable(emxArray_real32_T *cpu_a, emxArray_real32_T *gpu_a,
                  boolean_T *a_outdatedOnGpu, emxArray_real32_T *cpu_b,
                  emxArray_real32_T *gpu_b, boolean_T *b_outdatedOnGpu,
                  emxArray_real32_T *cpu_c, boolean_T *c_outdatedOnCpu,
                  emxArray_real32_T *gpu_c, boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real32_T b_gpu_c;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *b_cpu_c;
  emxArray_real32_T *cpu_expanded;
  int32_T k;
  int32_T mc;
  int32_T nc;
  boolean_T validLaunchParams;
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_real32_T(&gpu_expanded);
  gpuEmxReset_real32_T(&b_gpu_c);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
    mc = cpu_a->size[0] + cpu_b->size[0];
  } else {
    mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
  }
  if (cpu_a->size[1] == 0) {
    nc = 1;
  } else {
    nc = cpu_a->size[1];
  }
  k = cpu_c->size[0] * cpu_c->size[1];
  cpu_c->size[0] = mc;
  cpu_c->size[1] = nc;
  emxEnsureCapacity_real32_T(cpu_c, k, &yf_emlrtRTEI);
  k = mc * nc - 1;
  mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
  gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
  validLaunchParams = mwValidateLaunchParameters(grid, block);
  if (validLaunchParams) {
    conv2NonSeparable_kernel59<<<grid, block>>>(k, *gpu_c);
  }
  emxInit_real32_T(&b_cpu_c, 1, &gg_emlrtRTEI, true);
  emxInit_real32_T(&cpu_expanded, 2, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &bg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0) || (cpu_b->size[0] == 0)) {
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_real32_T(cpu_c, k, &yf_emlrtRTEI);
    k = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel60<<<grid, block>>>(k, *gpu_c);
    }
  } else if ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1) &&
             (cpu_b->size[0] == 1)) {
    real32_T alpha1;
    real32_T beta1;
    k = b_cpu_c->size[0];
    b_cpu_c->size[0] = 1;
    emxEnsureCapacity_real32_T(b_cpu_c, k, &fg_emlrtRTEI);
    alpha1 = 1.0F;
    beta1 = 0.0F;
    gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    gpuEmxEnsureCapacity_real32_T(b_cpu_c, &b_gpu_c, true);
    if (*a_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    cublasCheck(hipblasSgemm(getCublasGlobalHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
                            1, 1, 1, (float *)&alpha1, (float *)&gpu_a->data[0],
                            1, (float *)&gpu_b->data[0], 1, (float *)&beta1,
                            (float *)&b_gpu_c.data[0], 1),
                __FILE__, __LINE__);
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_real32_T(cpu_c, k, &yf_emlrtRTEI);
    k = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel61<<<grid, block>>>(b_gpu_c, k, *gpu_c);
    }
  } else {
    real_T blockDims[3];
    int32_T iv[2];
    int32_T OW;
    int32_T b_OH;
    int32_T offsetH;
    uint32_T OH;
    int8_T threadDims[3];
    OH = (static_cast<uint32_T>(cpu_a->size[0]) +
          static_cast<uint32_T>(cpu_b->size[0])) -
         1U;
    mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
    nc = static_cast<int32_T>(
        std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
    blockDims[0] =
        std::floor((static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
                   static_cast<real_T>(mc));
    blockDims[1] =
        std::floor((static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                                        static_cast<uint32_T>(nc)) -
                    1.0) /
                   static_cast<real_T>(nc));
    threadDims[0] = static_cast<int8_T>(mc);
    threadDims[1] = static_cast<int8_T>(nc);
    OW = cpu_a->size[1];
    if (OH > 2147483647U) {
      OH = 2147483647U;
    }
    b_OH = static_cast<int32_T>(OH);
    offsetH = static_cast<int32_T>(
                  std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
              static_cast<int32_T>(std::floor(
                  (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
    if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
      mc = MAX_int32_T;
    } else {
      mc = static_cast<int32_T>(OH) + cpu_b->size[0];
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[0] = mc - 1;
    emxEnsureCapacity_real32_T(cpu_expanded, k, &bg_emlrtRTEI);
    if (cpu_a->size[1] > 2147483646) {
      mc = MAX_int32_T;
    } else {
      mc = cpu_a->size[1] + 1;
    }
    k = cpu_expanded->size[0] * cpu_expanded->size[1];
    cpu_expanded->size[1] = mc - 1;
    emxEnsureCapacity_real32_T(cpu_expanded, k, &bg_emlrtRTEI);
    k = ((static_cast<int32_T>(OH) + cpu_b->size[0]) - 1) * cpu_a->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel62<<<grid, block>>>(k, gpu_expanded);
    }
    nc = cpu_a->size[0];
    k = cpu_y->size[0] * cpu_y->size[1];
    cpu_y->size[0] = 1;
    cpu_y->size[1] = cpu_a->size[0];
    emxEnsureCapacity_int32_T(cpu_y, k, &cg_emlrtRTEI);
    cpu_y->data[0] = 1;
    mc = 1;
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_y->data[k + 1] = mc;
    }
    k = cpu_iv->size[0];
    cpu_iv->size[0] = cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv, k, &bg_emlrtRTEI);
    k = cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel63<<<grid, block>>>(offsetH, gpu_y, k, gpu_iv);
    }
    nc = cpu_a->size[1];
    k = b_cpu_y->size[0] * b_cpu_y->size[1];
    b_cpu_y->size[0] = 1;
    b_cpu_y->size[1] = cpu_a->size[1];
    emxEnsureCapacity_int32_T(b_cpu_y, k, &cg_emlrtRTEI);
    b_cpu_y->data[0] = 1;
    mc = 1;
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      b_cpu_y->data[k + 1] = mc;
    }
    k = cpu_iv1->size[0];
    cpu_iv1->size[0] = b_cpu_y->size[1];
    emxEnsureCapacity_int32_T(cpu_iv1, k, &bg_emlrtRTEI);
    k = b_cpu_y->size[1] - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
    gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel64<<<grid, block>>>(b_gpu_y, k, gpu_iv1);
    }
    iv[0] = cpu_iv->size[0];
    iv[1] = cpu_iv1->size[0];
    mwGetLaunchParameters1D(computeNumIters(iv[1] - 1, iv[0] - 1), &grid,
                            &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_a, gpu_a, !*a_outdatedOnGpu);
    if (*a_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_a, cpu_a);
    }
    *a_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel65<<<grid, block>>>(
          *gpu_a, iv[0], gpu_iv1, gpu_iv, iv[0] - 1, iv[1] - 1, gpu_expanded,
          cpu_expanded->size[0U]);
    }
    nc = cpu_b->size[0];
    k = cpu_rows->size[0] * cpu_rows->size[1];
    cpu_rows->size[0] = 1;
    cpu_rows->size[1] = cpu_b->size[0];
    emxEnsureCapacity_int32_T(cpu_rows, k, &cg_emlrtRTEI);
    cpu_rows->data[0] = 0;
    mc = 0;
    for (k = 0; k <= nc - 2; k++) {
      mc++;
      cpu_rows->data[k + 1] = mc;
    }
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = static_cast<int32_T>(OH);
    cpu_c->size[1] = cpu_a->size[1];
    emxEnsureCapacity_real32_T(cpu_c, k, &bg_emlrtRTEI);
    if (blockDims[0] < 4.294967296E+9) {
      OH = static_cast<uint32_T>(blockDims[0]);
    } else {
      OH = MAX_uint32_T;
    }
    mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                            dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                            dim3(static_cast<uint32_T>(threadDims[0]),
                                 static_cast<uint32_T>(threadDims[1]), 1U),
                            &grid, &block);
    gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
    gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
    gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
    gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
    if (*b_outdatedOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
    }
    *b_outdatedOnGpu = false;
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2NonSeparable_kernel66<<<grid, block>>>(
          gpu_expanded, gpu_rows, *gpu_b, b_OH - 1, OW - 1, *gpu_c,
          cpu_b->size[0U], cpu_expanded->size[0U], cpu_c->size[0U]);
    }
  }
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv1);
  emxFree_int32_T(&b_cpu_y);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&cpu_y);
  emxFree_real32_T(&cpu_expanded);
  emxFree_real32_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&b_gpu_c);
  gpuEmxFree_real32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_int32_T(&gpu_rows);
  *c_outdatedOnCpu = true;
  *c_outdatedOnGpu = false;
}

} // namespace coder
static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel121(
    const int32_T b, emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1].re = 0.0F;
    c.data[i1].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel122(
    const int32_T b, emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1].re = 0.0F;
    c.data[i1].im = 0.0F;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel123(
    const real32_T cv_re, creal32_T b_data[1])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0].im = 0.0F;
    b_data[0].re = cv_re;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel124(
    const emxArray_creal32_T a, const creal32_T b_data[1],
    emxArray_creal32_T b_a)
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 1) {
    real32_T f;
    b_a.data[0].re = 0.0F;
    b_a.data[0].im = 0.0F;
    f = b_data[0].re;
    b_a.data[0].re += a.data[0].re * f - a.data[0].im * 0.0F;
    b_a.data[0].im += a.data[0].re * 0.0F + a.data[0].im * f;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel125(
    const emxArray_creal32_T a, const int32_T iv, emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(iv);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    c.data[i1] = a.data[i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel126(
    const int32_T OH, emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    expanded.data[i1].re = 0.0F;
    expanded.data[i1].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel127(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    i1 = static_cast<int32_T>(idx);
    q1 = y.data[i1];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[i1] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel128(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    i1 = static_cast<int32_T>(idx);
    iv1.data[i1] = y.data[i1] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel129(
    const emxArray_creal32_T a, const int32_T iv_dim0,
    const emxArray_int32_T iv1, const emxArray_int32_T iv, const int32_T b_iv,
    const int32_T c_iv, emxArray_creal32_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_iv) + 1UL) *
                (static_cast<uint64_T>(b_iv) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    q1 = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_iv) + 1UL));
    i1 = static_cast<int32_T>((idx - static_cast<uint64_T>(q1)) /
                              (static_cast<uint64_T>(b_iv) + 1UL));
    expanded.data[iv.data[q1] + expanded_dim0 * iv1.data[i1]] =
        a.data[q1 + iv_dim0 * i1];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel130(
    const int32_T offsetH, const emxArray_creal32_T expanded, const int32_T k,
    const emxArray_int32_T rows, const int32_T b_rows, emxArray_creal32_T newIm,
    int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_rows);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i1;
    int32_T q1;
    i1 = static_cast<int32_T>(idx);
    q1 = rows.data[i1];
    if (q1 > 2147483646 - k) {
      q1 = MAX_int32_T;
    } else {
      q1 = (k + q1) + 1;
    }
    newIm.data[i1] = expanded.data[(q1 + expanded_dim0 * offsetH) - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel131(
    const real32_T cv_re, const int32_T offsetH, const int32_T k,
    emxArray_creal32_T c, int32_T c_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[k + c_dim0 * offsetH].re = cv_re;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2NonSeparable_kernel132(
    const real32_T cv_im, const int32_T offsetH, const int32_T k,
    emxArray_creal32_T c, int32_T c_dim0)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[k + c_dim0 * offsetH].im = cv_im;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel59(
    const int32_T b, emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    c.data[i2] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel60(
    const int32_T b, emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    c.data[i2] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel61(
    const emxArray_real32_T c, const int32_T b, emxArray_real32_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    b_c.data[i2] = c.data[i2];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel62(
    const int32_T OH, emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    expanded.data[i2] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel63(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    int32_T q1;
    i2 = static_cast<int32_T>(idx);
    q1 = y.data[i2];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[i2] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel64(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    i2 = static_cast<int32_T>(idx);
    iv1.data[i2] = y.data[i2] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel65(
    const emxArray_real32_T a, const int32_T iv_dim0,
    const emxArray_int32_T iv1, const emxArray_int32_T iv, const int32_T b_iv,
    const int32_T c_iv, emxArray_real32_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(c_iv) + 1UL) *
                (static_cast<uint64_T>(b_iv) + 1UL) -
            1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T i2;
    int32_T ocol;
    ocol = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_iv) + 1UL));
    i2 = static_cast<int32_T>((idx - static_cast<uint64_T>(ocol)) /
                              (static_cast<uint64_T>(b_iv) + 1UL));
    expanded.data[iv.data[ocol] + expanded_dim0 * iv1.data[i2]] =
        a.data[ocol + iv_dim0 * i2];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel66(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_real32_T b, const int32_T c, const int32_T d,
    emxArray_real32_T b_c, int32_T b_dim0, int32_T expanded_dim0,
    int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(d) + 1UL) * (static_cast<uint64_T>(c) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    int32_T orow;
    real32_T b_cv;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(c) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(c) + 1UL));
    b_cv = 0.0F;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[(q1 + expanded_dim0 * ocol) - 1] *
              b.data[(b_dim0 - m) - 1];
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel67(
    const int32_T b, emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel68(
    const int32_T OH, emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel69(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel70(
    const emxArray_real32_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv.data[orow]] = a.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2NonSeparable_kernel71(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_real32_T b, const int32_T c, emxArray_real32_T b_c,
    int32_T b_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    real32_T b_cv;
    orow = static_cast<int32_T>(idx);
    b_cv = 0.0F;
    for (int32_T m{0}; m < b_dim0; m++) {
      int32_T q1;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv += expanded.data[q1 - 1] * b.data[(b_dim0 - m) - 1];
    }
    b_c.data[orow] = b_cv;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel104(
    const emxArray_creal32_T a, const int32_T b_a, emxArray_creal32_T c_a,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c_a.data[qY] = a.data[a_dim0 * qY];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel105(const int32_T a,
                                                    emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0F;
    c.data[qY].im = 0.0F;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel106(
    const emxArray_creal32_T a, const real32_T b, real32_T *f1, real32_T *f)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    *f = b * a.data[0].re;
    *f1 = b * a.data[0].im;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel107(
    const real32_T *f, const real32_T *f1, emxArray_creal32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0].im = *f1;
    c.data[0].re = *f;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel108(
    const emxArray_creal32_T c, const int32_T b_c, emxArray_creal32_T c_c,
    int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c_c.data[c_dim0 * qY] = c.data[qY];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel109(const int32_T a,
                                                    emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[qY].re = 0.0F;
    expanded.data[qY].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel110(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    iv.data[qY] = y.data[qY] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel111(
    const emxArray_creal32_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[iv.data[qY]] = a.data[qY];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel112(
    const emxArray_creal32_T expanded, const real32_T b, const int32_T c,
    emxArray_creal32_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_c.data[orow].re = b * expanded.data[orow].re;
    b_c.data[orow].im = b * expanded.data[orow].im;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel113(const int32_T b,
                                                    emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0F;
    c.data[qY].im = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel114(const int32_T b,
                                                    emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    c.data[qY].re = 0.0F;
    c.data[qY].im = 0.0F;
  }
}

static __global__ __launch_bounds__(32,
                                    1) void conv2_kernel115(const real32_T f,
                                                            creal32_T b_data[1])
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    b_data[0].im = 0.0F;
    b_data[0].re = f;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel116(
    const emxArray_creal32_T a, const creal32_T b_data[1], emxArray_creal32_T c)
{
  int32_T k;
  k = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (k < 1) {
    creal32_T b;
    b.re = b_data[0].re * a.data[0].re - 0.0F * a.data[0].im;
    b.im = b_data[0].re * a.data[0].im + 0.0F * a.data[0].re;
    c.data[0] = b;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel117(const int32_T OH,
                                                    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[qY] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel118(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T q1;
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    q1 = y.data[qY];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[qY] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel119(
    const emxArray_real32_T b, const int32_T mc, const emxArray_int32_T iv1,
    const int32_T c, emxArray_real32_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T qY;
    qY = static_cast<int32_T>(idx);
    expanded.data[iv1.data[qY] + expanded_dim0 * mc] = b.data[qY];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel120(
    const emxArray_real32_T expanded, const emxArray_int32_T cols,
    const emxArray_int32_T rows, const emxArray_creal32_T a, const int32_T b,
    const int32_T c, emxArray_creal32_T b_c, int32_T a_dim0, int32_T a_dim1,
    int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    creal32_T b_cv;
    int32_T ocol;
    int32_T orow;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_cv.re = 0.0F;
    b_cv.im = 0.0F;
    for (int32_T n{0}; n < a_dim1; n++) {
      for (int32_T m{0}; m < a_dim0; m++) {
        creal32_T b_b;
        int32_T q1;
        int32_T qY;
        b_b = a.data[((a_dim0 - m) + a_dim0 * ((a_dim1 - n) - 1)) - 1];
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv.re += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] * b_b.re;
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv.im += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] * b_b.im;
      }
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel133(const int32_T b,
                                                    emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow].re = 0.0F;
    c.data[orow].im = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel134(const int32_T OH,
                                                    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel135(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel136(
    const emxArray_real32_T b, const emxArray_int32_T iv1, const int32_T c,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv1.data[orow]] = b.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel137(
    const emxArray_real32_T expanded, const emxArray_int32_T rows,
    const emxArray_creal32_T a, const int32_T b, emxArray_creal32_T c,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    creal32_T b_cv;
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_cv.re = 0.0F;
    b_cv.im = 0.0F;
    for (int32_T m{0}; m < a_dim0; m++) {
      creal32_T b_b;
      int32_T q1;
      b_b = a.data[(a_dim0 - m) - 1];
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv.re += expanded.data[q1 - 1] * b_b.re;
      q1 = rows.data[m];
      if (q1 > 2147483646 - orow) {
        q1 = MAX_int32_T;
      } else {
        q1 = (orow + q1) + 1;
      }
      b_cv.im += expanded.data[q1 - 1] * b_b.im;
    }
    c.data[orow] = b_cv;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel138(const int32_T b,
                                                    emxArray_creal32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    c.data[orow].re = 0.0F;
    c.data[orow].im = 0.0F;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel139(const int32_T OH,
                                                    emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[orow].re = 0.0F;
    expanded.data[orow].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel140(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = y.data[orow];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv.data[orow] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel141(
    const emxArray_creal32_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_creal32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    expanded.data[iv.data[orow]] = a.data[orow];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel142(
    const emxArray_creal32_T expanded, const int32_T offsetH,
    const emxArray_int32_T rows, const int32_T b_rows, emxArray_creal32_T newIm)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_rows);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    int32_T q1;
    orow = static_cast<int32_T>(idx);
    q1 = rows.data[orow];
    if (q1 > 2147483646 - offsetH) {
      q1 = MAX_int32_T;
    } else {
      q1 = (offsetH + q1) + 1;
    }
    newIm.data[orow] = expanded.data[q1 - 1];
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel143(
    const real32_T cv_im, const real32_T cv_re, const int32_T offsetH,
    emxArray_creal32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[offsetH].re = cv_re;
    c.data[offsetH].im = cv_im;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel44(
    const emxArray_real32_T a, const int32_T b_a, emxArray_real32_T c_a,
    int32_T a_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c_a.data[ocol] = a.data[a_dim0 * ocol];
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel45(const int32_T a,
                                                           emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0F;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel46(
    const real32_T b, const emxArray_real32_T a, emxArray_real32_T c)
{
  int32_T tmpIdx;
  tmpIdx = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (tmpIdx < 1) {
    c.data[0] = a.data[0] * b;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel47(
    const emxArray_real32_T c, const int32_T b_c, emxArray_real32_T c_c,
    int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c_c.data[c_dim0 * ocol] = c.data[ocol];
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel48(const int32_T a,
                                                   emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(a);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[ocol] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel49(
    const emxArray_int32_T y, const int32_T b_y, emxArray_int32_T iv)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    iv.data[ocol] = y.data[ocol] - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel50(
    const emxArray_real32_T a, const emxArray_int32_T iv, const int32_T b,
    emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[iv.data[ocol]] = a.data[ocol];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel51(
    const real32_T b, const emxArray_real32_T expanded, const int32_T c,
    emxArray_real32_T b_c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T orow;
    orow = static_cast<int32_T>(idx);
    b_c.data[orow] = expanded.data[orow] * b;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel52(const int32_T b,
                                                           emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024,
                                    1) void conv2_kernel53(const int32_T b,
                                                           emxArray_real32_T c)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    c.data[ocol] = 0.0F;
  }
}

static __global__ __launch_bounds__(32, 1) void conv2_kernel54(
    const emxArray_real32_T a, const emxArray_real32_T b, emxArray_real32_T c)
{
  int32_T k;
  k = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (k < 1) {
    real32_T b_cv;
    b_cv = b.data[0] * a.data[0];
    c.data[0] = b_cv;
  }
}

static __global__
    __launch_bounds__(1024, 1) void conv2_kernel55(const int32_T OH,
                                                   emxArray_real32_T expanded)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(OH);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[ocol] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel56(
    const int32_T offsetH, const emxArray_int32_T y, const int32_T b_y,
    emxArray_int32_T iv1)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    int32_T q1;
    ocol = static_cast<int32_T>(idx);
    q1 = y.data[ocol];
    if ((offsetH > 0) && (q1 > MAX_int32_T - offsetH)) {
      q1 = MAX_int32_T;
    } else {
      q1 += offsetH;
    }
    iv1.data[ocol] = q1 - 1;
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel57(
    const emxArray_real32_T b, const int32_T mc, const emxArray_int32_T iv1,
    const int32_T c, emxArray_real32_T expanded, int32_T expanded_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    ocol = static_cast<int32_T>(idx);
    expanded.data[iv1.data[ocol] + expanded_dim0 * mc] = b.data[ocol];
  }
}

static __global__ __launch_bounds__(1024, 1) void conv2_kernel58(
    const emxArray_real32_T expanded, const emxArray_int32_T cols,
    const emxArray_int32_T rows, const emxArray_real32_T a, const int32_T b,
    const int32_T c, emxArray_real32_T b_c, int32_T a_dim0, int32_T a_dim1,
    int32_T expanded_dim0, int32_T c_dim0)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(c) + 1UL) * (static_cast<uint64_T>(b) + 1UL) - 1UL;
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T ocol;
    int32_T orow;
    real32_T b_cv;
    orow = static_cast<int32_T>(idx % (static_cast<uint64_T>(b) + 1UL));
    ocol = static_cast<int32_T>((idx - static_cast<uint64_T>(orow)) /
                                (static_cast<uint64_T>(b) + 1UL));
    b_cv = 0.0F;
    for (int32_T n{0}; n < a_dim1; n++) {
      for (int32_T m{0}; m < a_dim0; m++) {
        int32_T q1;
        int32_T qY;
        q1 = rows.data[m];
        if (q1 > 2147483646 - orow) {
          qY = MAX_int32_T;
        } else {
          qY = (orow + q1) + 1;
        }
        q1 = cols.data[n];
        if (q1 > 2147483646 - ocol) {
          q1 = MAX_int32_T;
        } else {
          q1 = (ocol + q1) + 1;
        }
        b_cv += expanded.data[(qY + expanded_dim0 * (q1 - 1)) - 1] *
                a.data[((a_dim0 - m) + a_dim0 * ((a_dim1 - n) - 1)) - 1];
      }
    }
    b_c.data[orow + c_dim0 * ocol] = b_cv;
  }
}

static void cublasCheck(hipblasStatus_t errCode, const char_T *file,
                        uint32_T b_line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPBLAS_STATUS_SUCCESS) {
    cublasGetErrorName(errCode, &errName);
    cublasGetErrorString(errCode, &errString);
    raiseCudaError(errCode, file, b_line, errName, errString);
  }
}

//
//
namespace coder {
void b_conv2(const emxArray_real32_T *cpu_a, emxArray_real32_T *cpu_b,
             boolean_T *b_outdatedOnCpu, emxArray_real32_T *gpu_b,
             boolean_T *b_outdatedOnGpu, emxArray_real32_T *cpu_c,
             boolean_T *c_outdatedOnCpu, emxArray_real32_T *gpu_c,
             boolean_T *c_outdatedOnGpu)
{
  emxArray_real32_T gpu_a;
  boolean_T a_outdatedOnCpu;
  boolean_T a_outdatedOnGpu;
  gpuEmxReset_real32_T(&gpu_a);
  a_outdatedOnCpu = false;
  a_outdatedOnGpu = true;
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    int32_T i;
    i = cpu_c->size[0];
    cpu_c->size[0] = 1;
    emxEnsureCapacity_real32_T(cpu_c, i, &ag_emlrtRTEI);
    if (*b_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
    }
    *b_outdatedOnCpu = false;
    cpu_c->data[0] = cpu_a->data[0] * cpu_b->data[0];
    *c_outdatedOnCpu = false;
    *c_outdatedOnGpu = true;
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    b_conv2NonSeparable(cpu_b, b_outdatedOnCpu, gpu_b, b_outdatedOnGpu,
                        (emxArray_real32_T *)cpu_a, &a_outdatedOnCpu, &gpu_a,
                        &a_outdatedOnGpu, cpu_c, c_outdatedOnCpu, gpu_c,
                        c_outdatedOnGpu);
  } else {
    b_conv2NonSeparable((emxArray_real32_T *)cpu_a, &a_outdatedOnCpu, &gpu_a,
                        &a_outdatedOnGpu, cpu_b, b_outdatedOnCpu, gpu_b,
                        b_outdatedOnGpu, cpu_c, c_outdatedOnCpu, gpu_c,
                        c_outdatedOnGpu);
  }
  gpuEmxFree_real32_T(&gpu_a);
}

//
//
void c_conv2(const emxArray_creal32_T *cpu_a, emxArray_real32_T *cpu_b,
             boolean_T *b_outdatedOnCpu, emxArray_real32_T *gpu_b,
             boolean_T *b_outdatedOnGpu, emxArray_creal32_T *cpu_c,
             boolean_T *c_outdatedOnCpu, emxArray_creal32_T *gpu_c,
             boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal32_T b_gpu_a;
  emxArray_creal32_T b_gpu_c;
  emxArray_creal32_T gpu_a;
  emxArray_creal32_T gpu_expanded;
  emxArray_creal32_T *b_cpu_a;
  emxArray_creal32_T *b_cpu_c;
  emxArray_creal32_T *cpu_expanded;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_cols;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_cols;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real32_T b_gpu_expanded;
  emxArray_real32_T *b_cpu_expanded;
  creal32_T(*gpu_b_data)[1];
  real32_T *gpu_f;
  real32_T *gpu_f1;
  boolean_T a_outdatedOnGpu;
  checkCudaError(mwCudaMalloc(&gpu_f1, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_f, 4UL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_b_data, 8UL), __FILE__, __LINE__);
  gpuEmxReset_int32_T(&gpu_cols);
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&b_gpu_expanded);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_creal32_T(&gpu_expanded);
  gpuEmxReset_creal32_T(&b_gpu_a);
  gpuEmxReset_creal32_T(&b_gpu_c);
  gpuEmxReset_creal32_T(&gpu_a);
  a_outdatedOnGpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_creal32_T(&b_cpu_c, 1, &dg_emlrtRTEI, true);
  emxInit_creal32_T(&b_cpu_a, 1, &vf_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_expanded, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_expanded, 2, &bg_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_cols, 2, &bg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    if (cpu_a->size[1] < 1) {
      int32_T k;
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      emxEnsureCapacity_creal32_T(cpu_c, k, &xf_emlrtRTEI);
      cpu_c->data[0].im = 0.0F;
      cpu_c->data[0].re = 0.0F;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      int32_T k;
      real32_T cpu_f;
      boolean_T validLaunchParams;
      k = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[1];
      emxEnsureCapacity_creal32_T(b_cpu_a, k, &vf_emlrtRTEI);
      k = cpu_a->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_creal32_T(b_cpu_a, &b_gpu_a, true);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel104<<<grid, block>>>(gpu_a, k, b_gpu_a, cpu_a->size[0U]);
      }
      if (*b_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      cpu_f = cpu_b->data[0];
      k = b_cpu_c->size[0];
      b_cpu_c->size[0] = b_cpu_a->size[0];
      emxEnsureCapacity_creal32_T(b_cpu_c, k, &yf_emlrtRTEI);
      k = b_cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal32_T(b_cpu_c, &b_gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel105<<<grid, block>>>(k, b_gpu_c);
      }
      if (b_cpu_a->size[0] == 1) {
        cpu_f = cpu_b->data[0];
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = 1;
        emxEnsureCapacity_creal32_T(b_cpu_c, k, &yf_emlrtRTEI);
        conv2_kernel106<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b_gpu_a, cpu_f,
                                                                 gpu_f1, gpu_f);
        gpuEmxEnsureCapacity_creal32_T(b_cpu_c, &b_gpu_c, true);
        conv2_kernel107<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_f, gpu_f1,
                                                                 b_gpu_c);
      } else {
        int32_T b_OH;
        int32_T mc;
        int32_T nc;
        int32_T offsetH;
        int8_T threadDims[3];
        nc = static_cast<int32_T>(std::fmin(
            32.0, (static_cast<real_T>(b_cpu_a->size[0]) + 1.0) - 1.0));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(b_cpu_a->size[0]) +
                                 static_cast<uint32_T>(nc)) -
             1.0) /
            static_cast<real_T>(nc)));
        threadDims[0] = static_cast<int8_T>(nc);
        b_OH = b_cpu_a->size[0];
        if (b_cpu_a->size[0] > 2147483646) {
          mc = MAX_int32_T;
        } else {
          mc = b_cpu_a->size[0] + 1;
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        emxEnsureCapacity_creal32_T(cpu_expanded, k, &bg_emlrtRTEI);
        k = b_cpu_a->size[0] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_creal32_T(cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel109<<<grid, block>>>(k, gpu_expanded);
        }
        nc = b_cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = b_cpu_a->size[0];
        emxEnsureCapacity_int32_T(cpu_y, k, &cg_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        for (k = 0; k <= nc - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        emxEnsureCapacity_int32_T(cpu_iv, k, &bg_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel110<<<grid, block>>>(gpu_y, k, gpu_iv);
        }
        nc = cpu_iv->size[0];
        mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel111<<<grid, block>>>(b_gpu_a, gpu_iv, nc - 1,
                                           gpu_expanded);
        }
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = b_cpu_a->size[0];
        emxEnsureCapacity_creal32_T(b_cpu_c, k, &bg_emlrtRTEI);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims[0]), 1U, 1U), &grid, &block);
        gpuEmxEnsureCapacity_creal32_T(b_cpu_c, &b_gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel112<<<grid, block>>>(gpu_expanded, cpu_f, b_OH - 1,
                                           b_gpu_c);
        }
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = b_cpu_c->size[0];
      emxEnsureCapacity_creal32_T(cpu_c, k, &ag_emlrtRTEI);
      k = b_cpu_c->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel108<<<grid, block>>>(b_gpu_c, k, *gpu_c, cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    int32_T nc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    if (cpu_a->size[1] == 0) {
      nc = 1;
    } else {
      nc = cpu_a->size[1];
    }
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_creal32_T(cpu_c, k, &wf_emlrtRTEI);
    k = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2_kernel113<<<grid, block>>>(k, *gpu_c);
    }
    if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0)) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = nc;
      emxEnsureCapacity_creal32_T(cpu_c, k, &wf_emlrtRTEI);
      k = mc * nc - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel114<<<grid, block>>>(k, *gpu_c);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else if ((cpu_b->size[0] == 1) &&
               ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1))) {
      real32_T cpu_f;
      if (*b_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      cpu_f = cpu_b->data[0];
      conv2_kernel115<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(cpu_f,
                                                               *gpu_b_data);
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      emxEnsureCapacity_creal32_T(cpu_c, k, &wf_emlrtRTEI);
      gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
      conv2_kernel116<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          gpu_a, *gpu_b_data, *gpu_c);
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else {
      real_T blockDims[3];
      int32_T OW;
      int32_T b_OH;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims[3];
      OH = (static_cast<uint32_T>(cpu_b->size[0]) +
            static_cast<uint32_T>(cpu_a->size[0])) -
           1U;
      nc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      mc = static_cast<int32_T>(
          std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
      blockDims[0] = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(nc) - 1.0)) /
          static_cast<real_T>(nc));
      blockDims[1] = std::floor(
          (static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                               static_cast<uint32_T>(mc)) -
           1.0) /
          static_cast<real_T>(mc));
      threadDims[0] = static_cast<int8_T>(nc);
      threadDims[1] = static_cast<int8_T>(mc);
      OW = cpu_a->size[1];
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
      if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_a->size[0];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[0] = mc - 1;
      emxEnsureCapacity_real32_T(b_cpu_expanded, k, &bg_emlrtRTEI);
      if (cpu_a->size[1] > MAX_int32_T - cpu_a->size[1]) {
        mc = MAX_int32_T;
      } else {
        mc = cpu_a->size[1] + cpu_a->size[1];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[1] = mc - 1;
      emxEnsureCapacity_real32_T(b_cpu_expanded, k, &bg_emlrtRTEI);
      k = ((static_cast<int32_T>(OH) + cpu_a->size[0]) - 1) *
              ((cpu_a->size[1] + cpu_a->size[1]) - 1) -
          1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(b_cpu_expanded, &b_gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel117<<<grid, block>>>(k, b_gpu_expanded);
      }
      nc = cpu_b->size[0];
      k = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = 1;
      b_cpu_y->size[1] = cpu_b->size[0];
      emxEnsureCapacity_int32_T(b_cpu_y, k, &cg_emlrtRTEI);
      b_cpu_y->data[0] = 1;
      mc = 1;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        b_cpu_y->data[k + 1] = mc;
      }
      k = cpu_iv1->size[0];
      cpu_iv1->size[0] = b_cpu_y->size[1];
      emxEnsureCapacity_int32_T(cpu_iv1, k, &bg_emlrtRTEI);
      k = b_cpu_y->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel118<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv1);
      }
      mc = static_cast<int32_T>(
               std::floor(static_cast<real_T>(cpu_a->size[1]) / 2.0)) +
           static_cast<int32_T>(
               std::floor((static_cast<real_T>(cpu_a->size[1]) - 1.0) / 2.0));
      nc = cpu_iv1->size[0];
      mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel119<<<grid, block>>>(*gpu_b, mc, gpu_iv1, nc - 1,
                                         b_gpu_expanded,
                                         b_cpu_expanded->size[0U]);
      }
      nc = cpu_a->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_a->size[0];
      emxEnsureCapacity_int32_T(cpu_rows, k, &cg_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      nc = cpu_a->size[1];
      k = cpu_cols->size[0] * cpu_cols->size[1];
      cpu_cols->size[0] = 1;
      cpu_cols->size[1] = cpu_a->size[1];
      emxEnsureCapacity_int32_T(cpu_cols, k, &cg_emlrtRTEI);
      cpu_cols->data[0] = 0;
      mc = 0;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_cols->data[k + 1] = mc;
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      cpu_c->size[1] = cpu_a->size[1];
      emxEnsureCapacity_creal32_T(cpu_c, k, &bg_emlrtRTEI);
      if (blockDims[0] < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims[0]);
      } else {
        OH = MAX_uint32_T;
      }
      mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                              dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                              dim3(static_cast<uint32_T>(threadDims[0]),
                                   static_cast<uint32_T>(threadDims[1]), 1U),
                              &grid, &block);
      gpuEmxEnsureCapacity_int32_T(cpu_cols, &gpu_cols, false);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_cols, cpu_cols);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel120<<<grid, block>>>(
            b_gpu_expanded, gpu_cols, gpu_rows, gpu_a, b_OH - 1, OW - 1, *gpu_c,
            cpu_a->size[0U], cpu_a->size[1U], b_cpu_expanded->size[0U],
            cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else {
    if (*b_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
    }
    *b_outdatedOnCpu = false;
    c_conv2NonSeparable((emxArray_creal32_T *)cpu_a, &gpu_a, &a_outdatedOnGpu,
                        cpu_b, cpu_c, c_outdatedOnCpu, gpu_c, c_outdatedOnGpu);
  }
  emxFree_int32_T(&cpu_cols);
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv1);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&b_cpu_y);
  emxFree_real32_T(&b_cpu_expanded);
  emxFree_int32_T(&cpu_y);
  emxFree_creal32_T(&cpu_expanded);
  emxFree_creal32_T(&b_cpu_a);
  emxFree_creal32_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_creal32_T(&gpu_a);
  gpuEmxFree_creal32_T(&b_gpu_c);
  gpuEmxFree_creal32_T(&b_gpu_a);
  gpuEmxFree_creal32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_real32_T(&b_gpu_expanded);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_int32_T(&gpu_rows);
  gpuEmxFree_int32_T(&gpu_cols);
  checkCudaError(mwCudaFree(*gpu_b_data), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_f), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(gpu_f1), __FILE__, __LINE__);
}

//
//
void conv2(const emxArray_real32_T *cpu_a, emxArray_real32_T *cpu_b,
           boolean_T *b_outdatedOnCpu, emxArray_real32_T *gpu_b,
           boolean_T *b_outdatedOnGpu, emxArray_real32_T *cpu_c,
           boolean_T *c_outdatedOnCpu, emxArray_real32_T *gpu_c,
           boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_cols;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_cols;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real32_T b_gpu_a;
  emxArray_real32_T b_gpu_c;
  emxArray_real32_T b_gpu_expanded;
  emxArray_real32_T gpu_a;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *b_cpu_a;
  emxArray_real32_T *b_cpu_c;
  emxArray_real32_T *b_cpu_expanded;
  emxArray_real32_T *cpu_expanded;
  boolean_T a_outdatedOnGpu;
  gpuEmxReset_int32_T(&gpu_cols);
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&b_gpu_expanded);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_real32_T(&gpu_expanded);
  gpuEmxReset_real32_T(&b_gpu_a);
  gpuEmxReset_real32_T(&b_gpu_c);
  gpuEmxReset_real32_T(&gpu_a);
  a_outdatedOnGpu = true;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real32_T(&b_cpu_c, 1, &dg_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_a, 1, &vf_emlrtRTEI, true);
  emxInit_real32_T(&cpu_expanded, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_expanded, 2, &bg_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_cols, 2, &bg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    if (cpu_a->size[1] < 1) {
      int32_T k;
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      emxEnsureCapacity_real32_T(cpu_c, k, &xf_emlrtRTEI);
      cpu_c->data[0] = 0.0F;
      *c_outdatedOnCpu = false;
      *c_outdatedOnGpu = true;
    } else {
      int32_T k;
      real32_T b;
      boolean_T validLaunchParams;
      k = b_cpu_a->size[0];
      b_cpu_a->size[0] = cpu_a->size[1];
      emxEnsureCapacity_real32_T(b_cpu_a, k, &vf_emlrtRTEI);
      k = cpu_a->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_real32_T(b_cpu_a, &b_gpu_a, true);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel44<<<grid, block>>>(gpu_a, k, b_gpu_a, cpu_a->size[0U]);
      }
      if (*b_outdatedOnCpu) {
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
      }
      *b_outdatedOnCpu = false;
      b = cpu_b->data[0];
      k = b_cpu_c->size[0];
      b_cpu_c->size[0] = b_cpu_a->size[0];
      emxEnsureCapacity_real32_T(b_cpu_c, k, &yf_emlrtRTEI);
      k = b_cpu_a->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(b_cpu_c, &b_gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel45<<<grid, block>>>(k, b_gpu_c);
      }
      if (b_cpu_a->size[0] == 1) {
        b = cpu_b->data[0];
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = 1;
        emxEnsureCapacity_real32_T(b_cpu_c, k, &yf_emlrtRTEI);
        gpuEmxEnsureCapacity_real32_T(b_cpu_c, &b_gpu_c, true);
        conv2_kernel46<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(b, b_gpu_a,
                                                                b_gpu_c);
      } else {
        int32_T b_OH;
        int32_T mc;
        int32_T nc;
        int32_T offsetH;
        int8_T threadDims[3];
        nc = static_cast<int32_T>(std::fmin(
            32.0, (static_cast<real_T>(b_cpu_a->size[0]) + 1.0) - 1.0));
        offsetH = static_cast<int32_T>(std::floor(
            (static_cast<real_T>(static_cast<uint32_T>(b_cpu_a->size[0]) +
                                 static_cast<uint32_T>(nc)) -
             1.0) /
            static_cast<real_T>(nc)));
        threadDims[0] = static_cast<int8_T>(nc);
        b_OH = b_cpu_a->size[0];
        if (b_cpu_a->size[0] > 2147483646) {
          mc = MAX_int32_T;
        } else {
          mc = b_cpu_a->size[0] + 1;
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        emxEnsureCapacity_real32_T(cpu_expanded, k, &bg_emlrtRTEI);
        k = b_cpu_a->size[0] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_real32_T(cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel48<<<grid, block>>>(k, gpu_expanded);
        }
        nc = b_cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = b_cpu_a->size[0];
        emxEnsureCapacity_int32_T(cpu_y, k, &cg_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        for (k = 0; k <= nc - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        emxEnsureCapacity_int32_T(cpu_iv, k, &bg_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &gpu_y, false);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel49<<<grid, block>>>(gpu_y, k, gpu_iv);
        }
        nc = cpu_iv->size[0];
        mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                                2147483647U);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel50<<<grid, block>>>(b_gpu_a, gpu_iv, nc - 1,
                                          gpu_expanded);
        }
        k = b_cpu_c->size[0];
        b_cpu_c->size[0] = b_cpu_a->size[0];
        emxEnsureCapacity_real32_T(b_cpu_c, k, &bg_emlrtRTEI);
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1),
            dim3(static_cast<uint32_T>(offsetH), 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims[0]), 1U, 1U), &grid, &block);
        gpuEmxEnsureCapacity_real32_T(b_cpu_c, &b_gpu_c, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel51<<<grid, block>>>(b, gpu_expanded, b_OH - 1, b_gpu_c);
        }
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = b_cpu_c->size[0];
      emxEnsureCapacity_real32_T(cpu_c, k, &ag_emlrtRTEI);
      k = b_cpu_c->size[0] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel47<<<grid, block>>>(b_gpu_c, k, *gpu_c, cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    int32_T nc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    if (cpu_a->size[1] == 0) {
      nc = 1;
    } else {
      nc = cpu_a->size[1];
    }
    k = cpu_c->size[0] * cpu_c->size[1];
    cpu_c->size[0] = mc;
    cpu_c->size[1] = nc;
    emxEnsureCapacity_real32_T(cpu_c, k, &wf_emlrtRTEI);
    k = mc * nc - 1;
    mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
    gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2_kernel52<<<grid, block>>>(k, *gpu_c);
    }
    if ((cpu_a->size[0] == 0) || (cpu_a->size[1] == 0)) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = mc;
      cpu_c->size[1] = nc;
      emxEnsureCapacity_real32_T(cpu_c, k, &wf_emlrtRTEI);
      k = mc * nc - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel53<<<grid, block>>>(k, *gpu_c);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else if ((cpu_b->size[0] == 1) &&
               ((cpu_a->size[0] == 1) && (cpu_a->size[1] == 1))) {
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = 1;
      cpu_c->size[1] = 1;
      emxEnsureCapacity_real32_T(cpu_c, k, &wf_emlrtRTEI);
      gpuEmxEnsureCapacity_real32_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_a, cpu_a);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      conv2_kernel54<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(gpu_a, *gpu_b,
                                                              *gpu_c);
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    } else {
      real_T blockDims[3];
      int32_T OW;
      int32_T b_OH;
      int32_T offsetH;
      uint32_T OH;
      int8_T threadDims[3];
      OH = (static_cast<uint32_T>(cpu_b->size[0]) +
            static_cast<uint32_T>(cpu_a->size[0])) -
           1U;
      nc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
      mc = static_cast<int32_T>(
          std::fmin(32.0, (static_cast<real_T>(cpu_a->size[1]) + 1.0) - 1.0));
      blockDims[0] = std::floor(
          (static_cast<real_T>(OH) + (static_cast<real_T>(nc) - 1.0)) /
          static_cast<real_T>(nc));
      blockDims[1] = std::floor(
          (static_cast<real_T>(static_cast<uint32_T>(cpu_a->size[1]) +
                               static_cast<uint32_T>(mc)) -
           1.0) /
          static_cast<real_T>(mc));
      threadDims[0] = static_cast<int8_T>(nc);
      threadDims[1] = static_cast<int8_T>(mc);
      OW = cpu_a->size[1];
      if (OH > 2147483647U) {
        OH = 2147483647U;
      }
      b_OH = static_cast<int32_T>(OH);
      offsetH = static_cast<int32_T>(
                    std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                static_cast<int32_T>(std::floor(
                    (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
      if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
        mc = MAX_int32_T;
      } else {
        mc = static_cast<int32_T>(OH) + cpu_a->size[0];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[0] = mc - 1;
      emxEnsureCapacity_real32_T(b_cpu_expanded, k, &bg_emlrtRTEI);
      if (cpu_a->size[1] > MAX_int32_T - cpu_a->size[1]) {
        mc = MAX_int32_T;
      } else {
        mc = cpu_a->size[1] + cpu_a->size[1];
      }
      k = b_cpu_expanded->size[0] * b_cpu_expanded->size[1];
      b_cpu_expanded->size[1] = mc - 1;
      emxEnsureCapacity_real32_T(b_cpu_expanded, k, &bg_emlrtRTEI);
      k = ((static_cast<int32_T>(OH) + cpu_a->size[0]) - 1) *
              ((cpu_a->size[1] + cpu_a->size[1]) - 1) -
          1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_real32_T(b_cpu_expanded, &b_gpu_expanded, true);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel55<<<grid, block>>>(k, b_gpu_expanded);
      }
      nc = cpu_b->size[0];
      k = b_cpu_y->size[0] * b_cpu_y->size[1];
      b_cpu_y->size[0] = 1;
      b_cpu_y->size[1] = cpu_b->size[0];
      emxEnsureCapacity_int32_T(b_cpu_y, k, &cg_emlrtRTEI);
      b_cpu_y->data[0] = 1;
      mc = 1;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        b_cpu_y->data[k + 1] = mc;
      }
      k = cpu_iv1->size[0];
      cpu_iv1->size[0] = b_cpu_y->size[1];
      emxEnsureCapacity_int32_T(cpu_iv1, k, &bg_emlrtRTEI);
      k = b_cpu_y->size[1] - 1;
      mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
      gpuEmxEnsureCapacity_int32_T(b_cpu_y, &b_gpu_y, false);
      gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, b_cpu_y);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel56<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv1);
      }
      mc = static_cast<int32_T>(
               std::floor(static_cast<real_T>(cpu_a->size[1]) / 2.0)) +
           static_cast<int32_T>(
               std::floor((static_cast<real_T>(cpu_a->size[1]) - 1.0) / 2.0));
      nc = cpu_iv1->size[0];
      mwGetLaunchParameters1D(computeNumIters(nc - 1), &grid, &block,
                              2147483647U);
      gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
      if (*b_outdatedOnGpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
      }
      *b_outdatedOnGpu = false;
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel57<<<grid, block>>>(*gpu_b, mc, gpu_iv1, nc - 1,
                                        b_gpu_expanded,
                                        b_cpu_expanded->size[0U]);
      }
      nc = cpu_a->size[0];
      k = cpu_rows->size[0] * cpu_rows->size[1];
      cpu_rows->size[0] = 1;
      cpu_rows->size[1] = cpu_a->size[0];
      emxEnsureCapacity_int32_T(cpu_rows, k, &cg_emlrtRTEI);
      cpu_rows->data[0] = 0;
      mc = 0;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_rows->data[k + 1] = mc;
      }
      nc = cpu_a->size[1];
      k = cpu_cols->size[0] * cpu_cols->size[1];
      cpu_cols->size[0] = 1;
      cpu_cols->size[1] = cpu_a->size[1];
      emxEnsureCapacity_int32_T(cpu_cols, k, &cg_emlrtRTEI);
      cpu_cols->data[0] = 0;
      mc = 0;
      for (k = 0; k <= nc - 2; k++) {
        mc++;
        cpu_cols->data[k + 1] = mc;
      }
      k = cpu_c->size[0] * cpu_c->size[1];
      cpu_c->size[0] = static_cast<int32_T>(OH);
      cpu_c->size[1] = cpu_a->size[1];
      emxEnsureCapacity_real32_T(cpu_c, k, &bg_emlrtRTEI);
      if (blockDims[0] < 4.294967296E+9) {
        OH = static_cast<uint32_T>(blockDims[0]);
      } else {
        OH = MAX_uint32_T;
      }
      mwApplyLaunchParameters(computeNumIters(OW - 1, b_OH - 1),
                              dim3(OH, static_cast<uint32_T>(blockDims[1]), 1U),
                              dim3(static_cast<uint32_T>(threadDims[0]),
                                   static_cast<uint32_T>(threadDims[1]), 1U),
                              &grid, &block);
      gpuEmxEnsureCapacity_int32_T(cpu_cols, &gpu_cols, false);
      gpuEmxEnsureCapacity_int32_T(cpu_rows, &gpu_rows, false);
      gpuEmxEnsureCapacity_real32_T(cpu_a, &gpu_a, false);
      gpuEmxEnsureCapacity_real32_T(cpu_c, gpu_c, true);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_cols, cpu_cols);
      gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, cpu_rows);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_a, cpu_a);
      validLaunchParams = mwValidateLaunchParameters(grid, block);
      if (validLaunchParams) {
        conv2_kernel58<<<grid, block>>>(
            b_gpu_expanded, gpu_cols, gpu_rows, gpu_a, b_OH - 1, OW - 1, *gpu_c,
            cpu_a->size[0U], cpu_a->size[1U], b_cpu_expanded->size[0U],
            cpu_c->size[0U]);
      }
      *c_outdatedOnGpu = false;
      *c_outdatedOnCpu = true;
    }
  } else {
    conv2NonSeparable((emxArray_real32_T *)cpu_a, &gpu_a, &a_outdatedOnGpu,
                      cpu_b, gpu_b, b_outdatedOnGpu, cpu_c, c_outdatedOnCpu,
                      gpu_c, c_outdatedOnGpu);
  }
  emxFree_int32_T(&cpu_cols);
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv1);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&b_cpu_y);
  emxFree_real32_T(&b_cpu_expanded);
  emxFree_int32_T(&cpu_y);
  emxFree_real32_T(&cpu_expanded);
  emxFree_real32_T(&b_cpu_a);
  emxFree_real32_T(&b_cpu_c);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_a);
  gpuEmxFree_real32_T(&b_gpu_c);
  gpuEmxFree_real32_T(&b_gpu_a);
  gpuEmxFree_real32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_real32_T(&b_gpu_expanded);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_int32_T(&gpu_rows);
  gpuEmxFree_int32_T(&gpu_cols);
}

//
//
void d_conv2(const emxArray_creal32_T *cpu_a, emxArray_real32_T *cpu_b,
             boolean_T *b_outdatedOnCpu, emxArray_real32_T *gpu_b,
             boolean_T *b_outdatedOnGpu, emxArray_creal32_T *cpu_c,
             boolean_T *c_outdatedOnCpu, emxArray_creal32_T *gpu_c,
             boolean_T *c_outdatedOnGpu)
{
  dim3 block;
  dim3 grid;
  emxArray_creal32_T b_gpu_expanded;
  emxArray_creal32_T gpu_a;
  emxArray_creal32_T gpu_newIm;
  emxArray_creal32_T *cpu_expanded;
  emxArray_creal32_T *cpu_newIm;
  emxArray_int32_T b_gpu_rows;
  emxArray_int32_T b_gpu_y;
  emxArray_int32_T gpu_iv;
  emxArray_int32_T gpu_iv1;
  emxArray_int32_T gpu_rows;
  emxArray_int32_T gpu_y;
  emxArray_int32_T *b_cpu_rows;
  emxArray_int32_T *b_cpu_y;
  emxArray_int32_T *cpu_iv;
  emxArray_int32_T *cpu_iv1;
  emxArray_int32_T *cpu_rows;
  emxArray_int32_T *cpu_y;
  emxArray_real32_T gpu_expanded;
  emxArray_real32_T *b_cpu_expanded;
  gpuEmxReset_creal32_T(&gpu_newIm);
  gpuEmxReset_int32_T(&gpu_rows);
  gpuEmxReset_int32_T(&b_gpu_rows);
  gpuEmxReset_int32_T(&gpu_iv1);
  gpuEmxReset_int32_T(&gpu_iv);
  gpuEmxReset_int32_T(&gpu_y);
  gpuEmxReset_int32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&gpu_expanded);
  gpuEmxReset_creal32_T(&b_gpu_expanded);
  gpuEmxReset_creal32_T(&gpu_a);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_creal32_T(&cpu_expanded, 1, &bg_emlrtRTEI, true);
  emxInit_real32_T(&b_cpu_expanded, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_y, 2, &eg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_iv1, 1, &bg_emlrtRTEI, true);
  emxInit_int32_T(&cpu_rows, 2, &bg_emlrtRTEI, true);
  emxInit_int32_T(&b_cpu_rows, 2, &bg_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_newIm, 1, &bg_emlrtRTEI, true);
  if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
    int32_T k;
    k = cpu_c->size[0];
    cpu_c->size[0] = 1;
    emxEnsureCapacity_creal32_T(cpu_c, k, &ag_emlrtRTEI);
    if (*b_outdatedOnCpu) {
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
    }
    cpu_c->data[0].re = cpu_b->data[0] * cpu_a->data[0].re;
    *b_outdatedOnCpu = false;
    cpu_c->data[0].im = cpu_b->data[0] * cpu_a->data[0].im;
    *c_outdatedOnCpu = false;
    *c_outdatedOnGpu = true;
  } else if (cpu_b->size[0] > cpu_a->size[0]) {
    int32_T k;
    int32_T mc;
    boolean_T validLaunchParams;
    if (cpu_a->size[0] == 0) {
      mc = cpu_b->size[0];
    } else {
      mc = (cpu_b->size[0] + cpu_a->size[0]) - 1;
    }
    k = cpu_c->size[0];
    cpu_c->size[0] = mc;
    emxEnsureCapacity_creal32_T(cpu_c, k, &wf_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2_kernel133<<<grid, block>>>(mc - 1, *gpu_c);
    }
    *c_outdatedOnGpu = false;
    *c_outdatedOnCpu = true;
    if (cpu_a->size[0] != 0) {
      if ((cpu_b->size[0] == 1) && (cpu_a->size[0] == 1)) {
        creal32_T a;
        real32_T cv_im;
        real32_T cv_re;
        a = cpu_a->data[0];
        k = cpu_c->size[0];
        cpu_c->size[0] = 1;
        emxEnsureCapacity_creal32_T(cpu_c, k, &wf_emlrtRTEI);
        if (*b_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
        }
        cv_im = cpu_b->data[0] * a.re;
        *b_outdatedOnCpu = false;
        cv_re = cpu_b->data[0] * a.im;
        cpu_c->data[0].im = cv_re;
        cpu_c->data[0].re = cv_im;
        *c_outdatedOnCpu = false;
        *c_outdatedOnGpu = true;
      } else {
        real_T blockDims_idx_0;
        int32_T b_OH;
        int32_T n;
        int32_T offsetH;
        uint32_T OH;
        int8_T threadDims_idx_0;
        OH = (static_cast<uint32_T>(cpu_b->size[0]) +
              static_cast<uint32_T>(cpu_a->size[0])) -
             1U;
        mc = static_cast<int32_T>(std::fmin(32.0, static_cast<real_T>(OH)));
        blockDims_idx_0 = std::floor(
            (static_cast<real_T>(OH) + (static_cast<real_T>(mc) - 1.0)) /
            static_cast<real_T>(mc));
        threadDims_idx_0 = static_cast<int8_T>(mc);
        if (OH > 2147483647U) {
          OH = 2147483647U;
        }
        b_OH = static_cast<int32_T>(OH);
        offsetH = static_cast<int32_T>(
                      std::floor(static_cast<real_T>(cpu_a->size[0]) / 2.0)) +
                  static_cast<int32_T>(std::floor(
                      (static_cast<real_T>(cpu_a->size[0]) - 1.0) / 2.0));
        if (cpu_a->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
          mc = MAX_int32_T;
        } else {
          mc = static_cast<int32_T>(OH) + cpu_a->size[0];
        }
        k = b_cpu_expanded->size[0];
        b_cpu_expanded->size[0] = mc - 1;
        emxEnsureCapacity_real32_T(b_cpu_expanded, k, &bg_emlrtRTEI);
        k = (static_cast<int32_T>(OH) + cpu_a->size[0]) - 2;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_real32_T(b_cpu_expanded, &gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel134<<<grid, block>>>(k, gpu_expanded);
        }
        n = cpu_b->size[0];
        k = b_cpu_y->size[0] * b_cpu_y->size[1];
        b_cpu_y->size[0] = 1;
        b_cpu_y->size[1] = cpu_b->size[0];
        emxEnsureCapacity_int32_T(b_cpu_y, k, &cg_emlrtRTEI);
        b_cpu_y->data[0] = 1;
        mc = 1;
        for (k = 0; k <= n - 2; k++) {
          mc++;
          b_cpu_y->data[k + 1] = mc;
        }
        k = cpu_iv1->size[0];
        cpu_iv1->size[0] = b_cpu_y->size[1];
        emxEnsureCapacity_int32_T(cpu_iv1, k, &bg_emlrtRTEI);
        k = b_cpu_y->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_int32_T(b_cpu_y, &gpu_y, false);
        gpuEmxEnsureCapacity_int32_T(cpu_iv1, &gpu_iv1, true);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_y, b_cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel135<<<grid, block>>>(offsetH, gpu_y, k, gpu_iv1);
        }
        mc = cpu_iv1->size[0];
        mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_real32_T(cpu_b, gpu_b, !*b_outdatedOnGpu);
        if (*b_outdatedOnGpu) {
          gpuEmxMemcpyCpuToGpu_real32_T(gpu_b, cpu_b);
        }
        *b_outdatedOnGpu = false;
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel136<<<grid, block>>>(*gpu_b, gpu_iv1, mc - 1,
                                           gpu_expanded);
        }
        n = cpu_a->size[0];
        k = b_cpu_rows->size[0] * b_cpu_rows->size[1];
        b_cpu_rows->size[0] = 1;
        b_cpu_rows->size[1] = cpu_a->size[0];
        emxEnsureCapacity_int32_T(b_cpu_rows, k, &cg_emlrtRTEI);
        b_cpu_rows->data[0] = 0;
        mc = 0;
        for (k = 0; k <= n - 2; k++) {
          mc++;
          b_cpu_rows->data[k + 1] = mc;
        }
        k = cpu_c->size[0];
        cpu_c->size[0] = static_cast<int32_T>(OH);
        emxEnsureCapacity_creal32_T(cpu_c, k, &bg_emlrtRTEI);
        if (blockDims_idx_0 < 4.294967296E+9) {
          OH = static_cast<uint32_T>(blockDims_idx_0);
        } else {
          OH = MAX_uint32_T;
        }
        mwApplyLaunchParameters(
            computeNumIters(b_OH - 1), dim3(OH, 1U, 1U),
            dim3(static_cast<uint32_T>(threadDims_idx_0), 1U, 1U), &grid,
            &block);
        gpuEmxEnsureCapacity_int32_T(b_cpu_rows, &gpu_rows, false);
        gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
        gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
        gpuEmxMemcpyCpuToGpu_int32_T(&gpu_rows, b_cpu_rows);
        gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel137<<<grid, block>>>(gpu_expanded, gpu_rows, gpu_a,
                                           b_OH - 1, *gpu_c, cpu_a->size[0U]);
        }
      }
    }
  } else {
    int32_T k;
    int32_T mc;
    boolean_T validLaunchParams;
    if ((cpu_a->size[0] == 0) || (cpu_b->size[0] == 0)) {
      mc = cpu_a->size[0] + cpu_b->size[0];
    } else {
      mc = (cpu_a->size[0] + cpu_b->size[0]) - 1;
    }
    k = cpu_c->size[0];
    cpu_c->size[0] = mc;
    emxEnsureCapacity_creal32_T(cpu_c, k, &jg_emlrtRTEI);
    mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                            2147483647U);
    gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      conv2_kernel138<<<grid, block>>>(mc - 1, *gpu_c);
    }
    *c_outdatedOnGpu = false;
    *c_outdatedOnCpu = true;
    if ((cpu_a->size[0] != 0) && (cpu_b->size[0] != 0)) {
      if ((cpu_a->size[0] == 1) && (cpu_b->size[0] == 1)) {
        real32_T cv_im;
        real32_T cv_re;
        if (*b_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
        }
        *b_outdatedOnCpu = false;
        cv_re = cpu_b->data[0];
        k = cpu_c->size[0];
        cpu_c->size[0] = 1;
        emxEnsureCapacity_creal32_T(cpu_c, k, &jg_emlrtRTEI);
        cv_im = cv_re * cpu_a->data[0].re;
        cv_re *= cpu_a->data[0].im;
        cpu_c->data[0].im = cv_re;
        cpu_c->data[0].re = cv_im;
        *c_outdatedOnCpu = false;
        *c_outdatedOnGpu = true;
      } else {
        int32_T b_OH;
        int32_T n;
        int32_T offsetH;
        uint32_T OH;
        boolean_T c_needsGpuEnsureCapacity;
        boolean_T rows_needsGpuEnsureCapacity;
        boolean_T rows_outdatedOnGpu;
        OH = (static_cast<uint32_T>(cpu_a->size[0]) +
              static_cast<uint32_T>(cpu_b->size[0])) -
             1U;
        if (OH > 2147483647U) {
          OH = 2147483647U;
        }
        b_OH = static_cast<int32_T>(OH);
        offsetH = static_cast<int32_T>(
                      std::floor(static_cast<real_T>(cpu_b->size[0]) / 2.0)) +
                  static_cast<int32_T>(std::floor(
                      (static_cast<real_T>(cpu_b->size[0]) - 1.0) / 2.0));
        if (cpu_b->size[0] > MAX_int32_T - static_cast<int32_T>(OH)) {
          mc = MAX_int32_T;
        } else {
          mc = static_cast<int32_T>(OH) + cpu_b->size[0];
        }
        k = cpu_expanded->size[0];
        cpu_expanded->size[0] = mc - 1;
        emxEnsureCapacity_creal32_T(cpu_expanded, k, &bg_emlrtRTEI);
        k = (static_cast<int32_T>(OH) + cpu_b->size[0]) - 2;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_creal32_T(cpu_expanded, &b_gpu_expanded, true);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel139<<<grid, block>>>(k, b_gpu_expanded);
        }
        n = cpu_a->size[0];
        k = cpu_y->size[0] * cpu_y->size[1];
        cpu_y->size[0] = 1;
        cpu_y->size[1] = cpu_a->size[0];
        emxEnsureCapacity_int32_T(cpu_y, k, &cg_emlrtRTEI);
        cpu_y->data[0] = 1;
        mc = 1;
        for (k = 0; k <= n - 2; k++) {
          mc++;
          cpu_y->data[k + 1] = mc;
        }
        k = cpu_iv->size[0];
        cpu_iv->size[0] = cpu_y->size[1];
        emxEnsureCapacity_int32_T(cpu_iv, k, &bg_emlrtRTEI);
        k = cpu_y->size[1] - 1;
        mwGetLaunchParameters1D(computeNumIters(k), &grid, &block, 2147483647U);
        gpuEmxEnsureCapacity_int32_T(cpu_y, &b_gpu_y, false);
        gpuEmxEnsureCapacity_int32_T(cpu_iv, &gpu_iv, true);
        gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_y, cpu_y);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel140<<<grid, block>>>(offsetH, b_gpu_y, k, gpu_iv);
        }
        mc = cpu_iv->size[0];
        mwGetLaunchParameters1D(computeNumIters(mc - 1), &grid, &block,
                                2147483647U);
        gpuEmxEnsureCapacity_creal32_T(cpu_a, &gpu_a, false);
        gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_a, cpu_a);
        validLaunchParams = mwValidateLaunchParameters(grid, block);
        if (validLaunchParams) {
          conv2_kernel141<<<grid, block>>>(gpu_a, gpu_iv, mc - 1,
                                           b_gpu_expanded);
        }
        n = cpu_b->size[0];
        k = cpu_rows->size[0] * cpu_rows->size[1];
        cpu_rows->size[0] = 1;
        cpu_rows->size[1] = cpu_b->size[0];
        emxEnsureCapacity_int32_T(cpu_rows, k, &cg_emlrtRTEI);
        cpu_rows->data[0] = 0;
        rows_outdatedOnGpu = true;
        rows_needsGpuEnsureCapacity = true;
        mc = 0;
        for (k = 0; k <= n - 2; k++) {
          mc++;
          cpu_rows->data[k + 1] = mc;
        }
        k = cpu_c->size[0];
        cpu_c->size[0] = static_cast<int32_T>(OH);
        emxEnsureCapacity_creal32_T(cpu_c, k, &bg_emlrtRTEI);
        c_needsGpuEnsureCapacity = true;
        mc = cpu_b->size[0];
        for (offsetH = 0; offsetH < b_OH; offsetH++) {
          real32_T cv_im;
          real32_T cv_re;
          boolean_T newIm_outdatedOnCpu;
          k = cpu_newIm->size[0];
          cpu_newIm->size[0] = cpu_rows->size[1];
          emxEnsureCapacity_creal32_T(cpu_newIm, k, &bg_emlrtRTEI);
          k = cpu_rows->size[1] - 1;
          mwGetLaunchParameters1D(computeNumIters(k), &grid, &block,
                                  2147483647U);
          if (rows_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_int32_T(cpu_rows, &b_gpu_rows,
                                         !rows_outdatedOnGpu);
          }
          rows_needsGpuEnsureCapacity = false;
          gpuEmxEnsureCapacity_creal32_T(cpu_newIm, &gpu_newIm, true);
          if (rows_outdatedOnGpu) {
            gpuEmxMemcpyCpuToGpu_int32_T(&b_gpu_rows, cpu_rows);
          }
          rows_outdatedOnGpu = false;
          validLaunchParams = mwValidateLaunchParameters(grid, block);
          if (validLaunchParams) {
            conv2_kernel142<<<grid, block>>>(b_gpu_expanded, offsetH,
                                             b_gpu_rows, k, gpu_newIm);
          }
          newIm_outdatedOnCpu = true;
          cv_re = 0.0F;
          cv_im = 0.0F;
          for (n = 0; n < mc; n++) {
            if (newIm_outdatedOnCpu) {
              gpuEmxMemcpyGpuToCpu_creal32_T(cpu_newIm, &gpu_newIm);
            }
            if (*b_outdatedOnCpu) {
              gpuEmxMemcpyGpuToCpu_real32_T(cpu_b, gpu_b);
            }
            cv_re +=
                cpu_newIm->data[n].re * cpu_b->data[(cpu_b->size[0] - n) - 1];
            newIm_outdatedOnCpu = false;
            *b_outdatedOnCpu = false;
            cv_im +=
                cpu_newIm->data[n].im * cpu_b->data[(cpu_b->size[0] - n) - 1];
          }
          if (c_needsGpuEnsureCapacity) {
            gpuEmxEnsureCapacity_creal32_T(cpu_c, gpu_c, true);
          }
          c_needsGpuEnsureCapacity = false;
          conv2_kernel143<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
              cv_im, cv_re, offsetH, *gpu_c);
        }
      }
    }
  }
  emxFree_creal32_T(&cpu_newIm);
  emxFree_int32_T(&b_cpu_rows);
  emxFree_int32_T(&cpu_rows);
  emxFree_int32_T(&cpu_iv1);
  emxFree_int32_T(&cpu_iv);
  emxFree_int32_T(&b_cpu_y);
  emxFree_int32_T(&cpu_y);
  emxFree_real32_T(&b_cpu_expanded);
  emxFree_creal32_T(&cpu_expanded);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_creal32_T(&gpu_a);
  gpuEmxFree_creal32_T(&b_gpu_expanded);
  gpuEmxFree_real32_T(&gpu_expanded);
  gpuEmxFree_int32_T(&b_gpu_y);
  gpuEmxFree_int32_T(&gpu_y);
  gpuEmxFree_int32_T(&gpu_iv);
  gpuEmxFree_int32_T(&gpu_iv1);
  gpuEmxFree_int32_T(&b_gpu_rows);
  gpuEmxFree_int32_T(&gpu_rows);
  gpuEmxFree_creal32_T(&gpu_newIm);
}

} // namespace coder

// End of code generation (conv2.cu)

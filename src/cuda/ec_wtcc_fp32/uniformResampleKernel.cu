#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// uniformResampleKernel.cu
//
// Code generation for function 'uniformResampleKernel'
//

// Include files
#include "uniformResampleKernel.h"
#include "ec_wtcc_data.h"
#include "ec_wtcc_emxutil.h"
#include "ec_wtcc_mexutil.h"
#include "ec_wtcc_types.h"
#include "rt_nonfinite.h"
#include "upfirdn.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo yg_emlrtRTEI{
    59,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo ah_emlrtRTEI{
    17,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo bh_emlrtRTEI{
    23,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

static emlrtRTEInfo ch_emlrtRTEI{
    55,                      // lineNo
    9,                       // colNo
    "uniformResampleKernel", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformResampleKernel.m" // pName
};

static emlrtRTEInfo dh_emlrtRTEI{
    13,                  // lineNo
    1,                   // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo eh_emlrtRTEI{
    14,                  // lineNo
    15,                  // colNo
    "uniformMatrixCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformMatrixCore.m" // pName
};

static emlrtRTEInfo fh_emlrtRTEI{
    16,                        // lineNo
    5,                         // colNo
    "uniformScalarVectorCore", // fName
    "/usr/local/MATLAB/R2024a/toolbox/signal/signal/+signal/+internal/"
    "+resample/uniformScalarVectorCore.m" // pName
};

// Function Definitions
//
//
namespace coder {
namespace b_signal {
namespace internal {
namespace resample {
void b_uniformResampleAlongFirstDim(
    emxArray_creal32_T *cpu_xIn, boolean_T *xIn_outdatedOnCpu,
    emxArray_creal32_T *gpu_xIn, boolean_T *xIn_outdatedOnGpu, real_T opts_p,
    real_T opts_q, emxArray_real32_T *cpu_opts_filterWithPadding,
    boolean_T *c_opts_filterWithPadding_outdat,
    emxArray_real32_T *gpu_opts_filterWithPadding,
    boolean_T *d_opts_filterWithPadding_outdat, real_T opts_filterDelay)
{
  emxArray_creal32_T gpu_yOut;
  emxArray_creal32_T gpu_yRow;
  emxArray_creal32_T *cpu_yOut;
  emxArray_creal32_T *cpu_yRow;
  emxArray_creal32_T *yRow;
  emxArray_real_T *b_y;
  emxArray_real_T *y;
  boolean_T yRow_outdatedOnCpu;
  boolean_T yRow_outdatedOnGpu;
  gpuEmxReset_creal32_T(&gpu_yRow);
  gpuEmxReset_creal32_T(&gpu_yOut);
  yRow_outdatedOnCpu = false;
  yRow_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_creal32_T(&cpu_yOut, 2, &dh_emlrtRTEI, true);
  emxInit_real_T(&y, 2, &eh_emlrtRTEI, true);
  emxInit_creal32_T(&cpu_yRow, 2, &fh_emlrtRTEI, true);
  emxInit_real_T(&b_y, 2, &eb_emlrtRTEI, true);
  emxInit_creal32_T(&yRow, 1, &bh_emlrtRTEI, true);
  if ((!(opts_p == 1.0)) || (!(opts_q == 1.0))) {
    if ((cpu_xIn->size[0] == 1) || (cpu_xIn->size[1] == 1)) {
      real_T b;
      int32_T b_i;
      int32_T i;
      int32_T u1;
      if ((cpu_xIn->size[0] == 0) || (cpu_xIn->size[1] == 0)) {
        u1 = 0;
      } else {
        i = cpu_xIn->size[0];
        u1 = cpu_xIn->size[1];
        if (i >= u1) {
          u1 = i;
        }
      }
      b = std::ceil(static_cast<real_T>(u1) * opts_p / opts_q);
      if ((cpu_xIn->size[0] == 1) && (cpu_xIn->size[1] == 1)) {
        b_upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu,
                  cpu_opts_filterWithPadding, c_opts_filterWithPadding_outdat,
                  gpu_opts_filterWithPadding, d_opts_filterWithPadding_outdat,
                  opts_p, opts_q, cpu_yRow, &yRow_outdatedOnCpu, &gpu_yRow,
                  &yRow_outdatedOnGpu);
        if ((cpu_yRow->size[0] == 0) || (cpu_yRow->size[1] == 0)) {
          u1 = 0;
        } else {
          i = cpu_yRow->size[0];
          u1 = cpu_yRow->size[1];
          if (i >= u1) {
            u1 = i;
          }
        }
        b_i = cpu_yRow->size[0] * cpu_yRow->size[1];
        cpu_yRow->size[0] = u1;
        cpu_yRow->size[1] = 1;
        emxEnsureCapacity_creal32_T(cpu_yRow, b_i, &ah_emlrtRTEI);
      } else {
        b_upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu,
                  cpu_opts_filterWithPadding, c_opts_filterWithPadding_outdat,
                  gpu_opts_filterWithPadding, d_opts_filterWithPadding_outdat,
                  opts_p, opts_q, cpu_yRow, &yRow_outdatedOnCpu, &gpu_yRow,
                  &yRow_outdatedOnGpu);
      }
      if (std::isnan(b)) {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = 1;
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        b_y->data[0] = rtNaN;
      } else if (b < 1.0) {
        b_y->size[0] = 1;
        b_y->size[1] = 0;
      } else {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          b_y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
      }
      i = static_cast<int32_T>(b);
      b_i = yRow->size[0];
      yRow->size[0] = static_cast<int32_T>(b);
      emxEnsureCapacity_creal32_T(yRow, b_i, &bh_emlrtRTEI);
      for (b_i = 0; b_i < i; b_i++) {
        if (yRow_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_creal32_T(cpu_yRow, &gpu_yRow);
        }
        yRow_outdatedOnCpu = false;
        yRow->data[b_i] =
            cpu_yRow
                ->data[static_cast<int32_T>(opts_filterDelay + b_y->data[b_i]) -
                       1];
      }
      i = static_cast<int32_T>(b);
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = static_cast<int32_T>(b);
      cpu_xIn->size[1] = 1;
      emxEnsureCapacity_creal32_T(cpu_xIn, b_i, &ch_emlrtRTEI);
      for (u1 = 0; u1 < i; u1++) {
        if (*xIn_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xIn, gpu_xIn);
        }
        cpu_xIn->data[u1] = yRow->data[u1];
        *xIn_outdatedOnCpu = false;
        *xIn_outdatedOnGpu = true;
      }
    } else {
      real_T b;
      int32_T b_i;
      int32_T i;
      b = std::ceil(static_cast<real_T>(cpu_xIn->size[0]) * opts_p / opts_q);
      b_upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
                c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
                d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yOut,
                &yRow_outdatedOnCpu, &gpu_yOut, &yRow_outdatedOnGpu);
      if (std::isnan(b)) {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = 1;
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        y->data[0] = rtNaN;
      } else if (b < 1.0) {
        y->size[0] = 1;
        y->size[1] = 0;
      } else {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
      }
      i = cpu_yOut->size[1];
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = y->size[1];
      cpu_xIn->size[1] = cpu_yOut->size[1];
      emxEnsureCapacity_creal32_T(cpu_xIn, b_i, &yg_emlrtRTEI);
      for (b_i = 0; b_i < i; b_i++) {
        for (int32_T u1{0}; u1 < y->size[1]; u1++) {
          if (*xIn_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xIn, gpu_xIn);
          }
          if (yRow_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_creal32_T(cpu_yOut, &gpu_yOut);
          }
          yRow_outdatedOnCpu = false;
          cpu_xIn->data[u1 + cpu_xIn->size[0] * b_i] =
              cpu_yOut
                  ->data[(static_cast<int32_T>(opts_filterDelay + y->data[u1]) +
                          cpu_yOut->size[0] * b_i) -
                         1];
          *xIn_outdatedOnCpu = false;
          *xIn_outdatedOnGpu = true;
        }
      }
    }
  }
  emxFree_creal32_T(&yRow);
  emxFree_real_T(&b_y);
  emxFree_creal32_T(&cpu_yRow);
  emxFree_real_T(&y);
  emxFree_creal32_T(&cpu_yOut);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_creal32_T(&gpu_yOut);
  gpuEmxFree_creal32_T(&gpu_yRow);
}

//
//
void uniformResampleAlongFirstDim(
    emxArray_real32_T *cpu_xIn, boolean_T *xIn_outdatedOnCpu,
    emxArray_real32_T *gpu_xIn, boolean_T *xIn_outdatedOnGpu, real_T opts_p,
    real_T opts_q, emxArray_real32_T *cpu_opts_filterWithPadding,
    boolean_T *c_opts_filterWithPadding_outdat,
    emxArray_real32_T *gpu_opts_filterWithPadding,
    boolean_T *d_opts_filterWithPadding_outdat, real_T opts_filterDelay)
{
  emxArray_real32_T gpu_yOut;
  emxArray_real32_T gpu_yRow;
  emxArray_real32_T *cpu_yOut;
  emxArray_real32_T *cpu_yRow;
  emxArray_real32_T *yRow;
  emxArray_real_T *b_y;
  emxArray_real_T *y;
  boolean_T yRow_outdatedOnCpu;
  boolean_T yRow_outdatedOnGpu;
  gpuEmxReset_real32_T(&gpu_yRow);
  gpuEmxReset_real32_T(&gpu_yOut);
  yRow_outdatedOnCpu = false;
  yRow_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_real32_T(&cpu_yOut, 2, &dh_emlrtRTEI, true);
  emxInit_real_T(&y, 2, &eh_emlrtRTEI, true);
  emxInit_real32_T(&cpu_yRow, 2, &fh_emlrtRTEI, true);
  emxInit_real_T(&b_y, 2, &eb_emlrtRTEI, true);
  emxInit_real32_T(&yRow, 1, &bh_emlrtRTEI, true);
  if ((!(opts_p == 1.0)) || (!(opts_q == 1.0))) {
    if ((cpu_xIn->size[0] == 1) || (cpu_xIn->size[1] == 1)) {
      real_T b;
      int32_T b_i;
      int32_T i;
      int32_T u1;
      if ((cpu_xIn->size[0] == 0) || (cpu_xIn->size[1] == 0)) {
        u1 = 0;
      } else {
        i = cpu_xIn->size[0];
        u1 = cpu_xIn->size[1];
        if (i >= u1) {
          u1 = i;
        }
      }
      b = std::ceil(static_cast<real_T>(u1) * opts_p / opts_q);
      if ((cpu_xIn->size[0] == 1) && (cpu_xIn->size[1] == 1)) {
        upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
                c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
                d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
        if ((cpu_yRow->size[0] == 0) || (cpu_yRow->size[1] == 0)) {
          u1 = 0;
        } else {
          i = cpu_yRow->size[0];
          u1 = cpu_yRow->size[1];
          if (i >= u1) {
            u1 = i;
          }
        }
        b_i = cpu_yRow->size[0] * cpu_yRow->size[1];
        cpu_yRow->size[0] = u1;
        cpu_yRow->size[1] = 1;
        emxEnsureCapacity_real32_T(cpu_yRow, b_i, &ah_emlrtRTEI);
      } else {
        upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
                c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
                d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yRow,
                &yRow_outdatedOnCpu, &gpu_yRow, &yRow_outdatedOnGpu);
      }
      if (std::isnan(b)) {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = 1;
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        b_y->data[0] = rtNaN;
      } else if (b < 1.0) {
        b_y->size[0] = 1;
        b_y->size[1] = 0;
      } else {
        b_i = b_y->size[0] * b_y->size[1];
        b_y->size[0] = 1;
        b_y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        emxEnsureCapacity_real_T(b_y, b_i, &n_emlrtRTEI);
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          b_y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
      }
      i = static_cast<int32_T>(b);
      b_i = yRow->size[0];
      yRow->size[0] = static_cast<int32_T>(b);
      emxEnsureCapacity_real32_T(yRow, b_i, &bh_emlrtRTEI);
      for (b_i = 0; b_i < i; b_i++) {
        if (yRow_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_yRow, &gpu_yRow);
        }
        yRow_outdatedOnCpu = false;
        yRow->data[b_i] =
            cpu_yRow
                ->data[static_cast<int32_T>(opts_filterDelay + b_y->data[b_i]) -
                       1];
      }
      i = static_cast<int32_T>(b);
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = static_cast<int32_T>(b);
      cpu_xIn->size[1] = 1;
      emxEnsureCapacity_real32_T(cpu_xIn, b_i, &ch_emlrtRTEI);
      for (u1 = 0; u1 < i; u1++) {
        if (*xIn_outdatedOnCpu) {
          gpuEmxMemcpyGpuToCpu_real32_T(cpu_xIn, gpu_xIn);
        }
        cpu_xIn->data[u1] = yRow->data[u1];
        *xIn_outdatedOnCpu = false;
        *xIn_outdatedOnGpu = true;
      }
    } else {
      real_T b;
      int32_T b_i;
      int32_T i;
      b = std::ceil(static_cast<real_T>(cpu_xIn->size[0]) * opts_p / opts_q);
      upfirdn(cpu_xIn, gpu_xIn, xIn_outdatedOnGpu, cpu_opts_filterWithPadding,
              c_opts_filterWithPadding_outdat, gpu_opts_filterWithPadding,
              d_opts_filterWithPadding_outdat, opts_p, opts_q, cpu_yOut,
              &yRow_outdatedOnCpu, &gpu_yOut, &yRow_outdatedOnGpu);
      if (std::isnan(b)) {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = 1;
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        y->data[0] = rtNaN;
      } else if (b < 1.0) {
        y->size[0] = 1;
        y->size[1] = 0;
      } else {
        b_i = y->size[0] * y->size[1];
        y->size[0] = 1;
        y->size[1] = static_cast<int32_T>(b - 1.0) + 1;
        emxEnsureCapacity_real_T(y, b_i, &n_emlrtRTEI);
        for (b_i = 0; b_i <= static_cast<int32_T>(b - 1.0); b_i++) {
          y->data[b_i] = static_cast<real_T>(b_i) + 1.0;
        }
      }
      i = cpu_yOut->size[1];
      b_i = cpu_xIn->size[0] * cpu_xIn->size[1];
      cpu_xIn->size[0] = y->size[1];
      cpu_xIn->size[1] = cpu_yOut->size[1];
      emxEnsureCapacity_real32_T(cpu_xIn, b_i, &yg_emlrtRTEI);
      for (b_i = 0; b_i < i; b_i++) {
        for (int32_T u1{0}; u1 < y->size[1]; u1++) {
          if (*xIn_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_xIn, gpu_xIn);
          }
          if (yRow_outdatedOnCpu) {
            gpuEmxMemcpyGpuToCpu_real32_T(cpu_yOut, &gpu_yOut);
          }
          yRow_outdatedOnCpu = false;
          cpu_xIn->data[u1 + cpu_xIn->size[0] * b_i] =
              cpu_yOut
                  ->data[(static_cast<int32_T>(opts_filterDelay + y->data[u1]) +
                          cpu_yOut->size[0] * b_i) -
                         1];
          *xIn_outdatedOnCpu = false;
          *xIn_outdatedOnGpu = true;
        }
      }
    }
  }
  emxFree_real32_T(&yRow);
  emxFree_real_T(&b_y);
  emxFree_real32_T(&cpu_yRow);
  emxFree_real_T(&y);
  emxFree_real32_T(&cpu_yOut);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_real32_T(&gpu_yOut);
  gpuEmxFree_real32_T(&gpu_yRow);
}

} // namespace resample
} // namespace internal
} // namespace b_signal
} // namespace coder

// End of code generation (uniformResampleKernel.cu)

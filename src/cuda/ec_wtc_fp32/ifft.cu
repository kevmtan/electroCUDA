//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ifft.cu
//
// Code generation for function 'ifft'
//

// Include files
#include "ifft.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "hipfft/hipfft.h"
#include "nvtx3/nvToolsExt.h"
#include <cstring>

// Variable Definitions
static emlrtRTEInfo wd_emlrtRTEI{
    83,                                                              // lineNo
    1,                                                               // colNo
    "ifft",                                                          // fName
    "/usr/local/MATLAB/R2024a/toolbox/eml/lib/matlab/datafun/ifft.m" // pName
};

// Function Definitions
//
//
namespace coder {
void ifft(const emxArray_creal32_T *x, emxArray_creal32_T *y)
{
  emxArray_creal32_T gpu_x;
  emxArray_creal32_T *cpu_x;
  nvtxRangePushA("#fcn#ifft#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_x);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  if ((x->size[0] == 0) || (x->size[1] == 0)) {
    int32_T inembed;
    uint32_T ySize_idx_0;
    uint32_T ySize_idx_1;
    ySize_idx_0 = static_cast<uint32_T>(x->size[0]);
    ySize_idx_1 = static_cast<uint32_T>(x->size[1]);
    inembed = y->size[0] * y->size[1];
    y->size[0] = x->size[0];
    y->size[1] = x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(y, inembed, &wd_emlrtRTEI);
    if (static_cast<int32_T>(ySize_idx_0) * static_cast<int32_T>(ySize_idx_1) -
            1 >=
        0) {
      std::memset(&y->data[0], 0,
                  static_cast<uint32_T>(static_cast<int32_T>(ySize_idx_0) *
                                        static_cast<int32_T>(ySize_idx_1)) *
                      sizeof(creal32_T));
    }
  } else {
    hipfftHandle fftPlanHandle;
    int32_T inembed;
    real32_T x_re;
    boolean_T x_outdatedOnGpu;
    nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
    emxInit_creal32_T(&cpu_x, 2, &vd_emlrtRTEI, true);
    x_outdatedOnGpu = false;
    inembed = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = x->size[0];
    cpu_x->size[1] = x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_x, inembed, &vd_emlrtRTEI);
    profileLoopStart("ifft_loop_0", __LINE__, (x->size[0] * x->size[1] - 1) + 1,
                     "");
    for (inembed = 0; inembed < x->size[0] * x->size[1]; inembed++) {
      cpu_x->data[inembed] = x->data[inembed];
      x_outdatedOnGpu = true;
    }
    profileLoopEnd();
    inembed = x->size[1];
    fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, x->size[0], 1,
                                     HIPFFT_C2C, x->size[0]);
    nvtxMarkA("#gpuEmxEnsureCapacity_creal32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_creal32_T(cpu_x, &gpu_x, !x_outdatedOnGpu);
    if (x_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_x, cpu_x);
    }
    hipfftExecC2C(fftPlanHandle, (hipfftComplex *)&gpu_x.data[0],
                 (hipfftComplex *)&gpu_x.data[0], HIPFFT_BACKWARD);
    x_outdatedOnGpu = true;
    inembed = y->size[0] * y->size[1];
    y->size[0] = cpu_x->size[0];
    y->size[1] = cpu_x->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(y, inembed, &wd_emlrtRTEI);
    x_re = static_cast<real32_T>(x->size[1]);
    profileLoopStart("ifft_loop_1", __LINE__,
                     (cpu_x->size[0] * cpu_x->size[1] - 1) + 1, "");
    for (inembed = 0; inembed < cpu_x->size[0] * cpu_x->size[1]; inembed++) {
      real32_T ai;
      real32_T ar;
      if (x_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_creal32_T(cpu_x, &gpu_x);
      }
      ar = cpu_x->data[inembed].re;
      x_outdatedOnGpu = false;
      ai = cpu_x->data[inembed].im;
      if (ai == 0.0F) {
        y->data[inembed].re = ar / x_re;
        y->data[inembed].im = 0.0F;
      } else if (ar == 0.0F) {
        y->data[inembed].re = 0.0F;
        y->data[inembed].im = ai / x_re;
      } else {
        y->data[inembed].re = ar / x_re;
        y->data[inembed].im = ai / x_re;
      }
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
    emxFree_creal32_T(&cpu_x);
  }
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_x);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (ifft.cu)

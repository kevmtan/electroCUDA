#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// wcoherence.cu
//
// Code generation for function 'wcoherence'
//

// Include files
#include "wcoherence.h"
#include "bsxfun.h"
#include "cwtfilterbank.h"
#include "cwtfreqlimits.h"
#include "div.h"
#include "ec_wtc_fp_data.h"
#include "ec_wtc_fp_emxutil.h"
#include "ec_wtc_fp_mexutil.h"
#include "ec_wtc_fp_types.h"
#include "fft.h"
#include "ifft.h"
#include "log2.h"
#include "rt_nonfinite.h"
#include "smoothCFS.h"
#include "wt.h"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Variable Definitions
static emlrtRTEInfo fb_emlrtRTEI{
    187,                                                            // lineNo
    1,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo gb_emlrtRTEI{
    102,  // lineNo
    5,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo mb_emlrtRTEI{
    219,                                                            // lineNo
    1,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo ob_emlrtRTEI{
    170,                                                            // lineNo
    5,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo pb_emlrtRTEI{
    1,                                                              // lineNo
    37,                                                             // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo rb_emlrtRTEI{
    161,  // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/@cwtfilterbank/wt.m" // pName
};

static emlrtRTEInfo sb_emlrtRTEI{
    216,                                                            // lineNo
    39,                                                             // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo tb_emlrtRTEI{
    217,                                                            // lineNo
    39,                                                             // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

static emlrtRTEInfo ub_emlrtRTEI{
    220,                                                            // lineNo
    7,                                                              // colNo
    "wcoherence",                                                   // fName
    "/usr/local/MATLAB/R2024a/toolbox/wavelet/wavelet/wcoherence.m" // pName
};

// Function Declarations
static void binary_expand_op_1(emxArray_creal32_T *in1,
                               const emxArray_creal32_T *in2,
                               const emxArray_creal32_T *in3);

static real32_T rt_hypotf_snf(real32_T u0, real32_T u1);

static __global__ void wcoherence_kernel26(const int32_T i, const int32_T b,
                                           emxArray_real32_T x);

// Function Definitions
static void binary_expand_op_1(emxArray_creal32_T *in1,
                               const emxArray_creal32_T *in2,
                               const emxArray_creal32_T *in3)
{
  int32_T aux_0_1;
  int32_T aux_1_1;
  int32_T b;
  int32_T i;
  int32_T stride_0_0;
  int32_T stride_0_1;
  int32_T stride_1_0;
  int32_T stride_1_1;
  nvtxRangePushA("#fcn#binary_expand_op_1#" MW_AT_LOCATION);
  i = in1->size[0] * in1->size[1];
  if (in3->size[0] == 1) {
    in1->size[0] = in2->size[0];
  } else {
    in1->size[0] = in3->size[0];
  }
  if (in3->size[1] == 1) {
    in1->size[1] = in2->size[1];
  } else {
    in1->size[1] = in3->size[1];
  }
  nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
  emxEnsureCapacity_creal32_T(in1, i, &mb_emlrtRTEI);
  stride_0_0 = (in2->size[0] != 1);
  stride_0_1 = (in2->size[1] != 1);
  stride_1_0 = (in3->size[0] != 1);
  stride_1_1 = (in3->size[1] != 1);
  aux_0_1 = 0;
  aux_1_1 = 0;
  if (in3->size[1] == 1) {
    b = in2->size[1];
  } else {
    b = in3->size[1];
  }
  profileLoopStart("binary_expand_op_1_loop_0", __LINE__, (b - 1) + 1, "");
  for (i = 0; i < b; i++) {
    int32_T c;
    if (in3->size[0] == 1) {
      c = in2->size[0];
    } else {
      c = in3->size[0];
    }
    profileLoopStart("binary_expand_op_1_loop_1", __LINE__, (c - 1) + 1, "");
    for (int32_T i1{0}; i1 < c; i1++) {
      real32_T in3_im;
      real32_T in3_re;
      in3_re = in3->data[i1 * stride_1_0 + in3->size[0] * aux_1_1].re;
      in3_im = -in3->data[i1 * stride_1_0 + in3->size[0] * aux_1_1].im;
      in1->data[i1 + in1->size[0] * i].re =
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].re * in3_re -
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].im * in3_im;
      in1->data[i1 + in1->size[0] * i].im =
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].re * in3_im +
          in2->data[i1 * stride_0_0 + in2->size[0] * aux_0_1].im * in3_re;
    }
    profileLoopEnd();
    aux_1_1 += stride_1_1;
    aux_0_1 += stride_0_1;
  }
  profileLoopEnd();
  nvtxRangePop();
}

static real32_T rt_hypotf_snf(real32_T u0, real32_T u1)
{
  real32_T a;
  real32_T b;
  real32_T y;
  nvtxRangePushA("#fcn#rt_hypotf_snf#" MW_AT_LOCATION);
  a = std::abs(u0);
  b = std::abs(u1);
  if (a < b) {
    a /= b;
    y = b * std::sqrt(a * a + 1.0F);
  } else if (a > b) {
    b /= a;
    y = a * std::sqrt(b * b + 1.0F);
  } else if (std::isnan(b)) {
    y = rtNaNF;
  } else {
    y = a * 1.41421354F;
  }
  nvtxRangePop();
  return y;
}

static __global__ __launch_bounds__(1024, 1) void wcoherence_kernel26(
    const int32_T i, const int32_T b, emxArray_real32_T x)
{
  uint64_T gStride;
  uint64_T gThreadId;
  uint64_T loopEnd;
  gThreadId = mwGetGlobalThreadIndex();
  gStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b);
  for (uint64_T idx{gThreadId}; idx <= loopEnd; idx += gStride) {
    int32_T b_j1;
    int32_T j2;
    real32_T xtmp;
    b_j1 = static_cast<int32_T>(idx);
    j2 = i - b_j1;
    xtmp = x.data[b_j1];
    x.data[b_j1] = x.data[j2 - 1];
    x.data[j2 - 1] = xtmp;
  }
}

//
//
namespace coder {
void wcoherence(const emxArray_real32_T *x, const emxArray_real32_T *y,
                real_T varargin_1, const real_T varargin_3[2],
                real_T varargin_5, emxArray_real32_T *wcoh,
                emxArray_creal32_T *cpu_wcs, boolean_T *wcs_outdatedOnCpu,
                emxArray_creal32_T *gpu_wcs, boolean_T *wcs_outdatedOnGpu)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 7};
  static const char_T rfmt[7]{'%', '2', '3', '.', '1', '5', 'e'};
  cwtfilterbank fb;
  dim3 block;
  dim3 grid;
  emxArray_creal32_T gpu_xposdft;
  emxArray_creal32_T *cfspos;
  emxArray_creal32_T *cpu_xposdft;
  emxArray_creal32_T *cwtx;
  emxArray_creal32_T *r;
  emxArray_real32_T b_gpu_y;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_xv;
  emxArray_real32_T gpu_y;
  emxArray_real32_T *b_cpu_y;
  emxArray_real32_T *coitmp;
  emxArray_real32_T *cpu_x;
  emxArray_real32_T *cpu_xv;
  emxArray_real32_T *cpu_y;
  emxArray_real32_T *f;
  emxArray_real32_T *g_y;
  emxArray_real32_T *i_y;
  emxArray_real32_T *j_y;
  emxArray_real32_T *k_y;
  emxArray_real32_T *psihat;
  emxArray_real_T gpu_scales;
  emxArray_real_T *cpu_scales;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *f_y;
  const mxArray *h_y;
  const mxArray *m;
  const mxArray *m1;
  const mxArray *m2;
  const mxArray *m3;
  real_T FLimits[2];
  real_T Norig;
  real_T b_y;
  real_T c_y;
  real_T maxperiod;
  real_T minperiod;
  real_T ns;
  int32_T b_i;
  int32_T i;
  int32_T i3;
  int32_T i5;
  int32_T k;
  int32_T nd2;
  char_T unusedExpr[23];
  boolean_T exitg1;
  boolean_T guard1;
  boolean_T scales_outdatedOnGpu;
  boolean_T xposdft_outdatedOnCpu;
  boolean_T xposdft_outdatedOnGpu;
  boolean_T xv_outdatedOnCpu;
  boolean_T xv_outdatedOnGpu;
  nvtxRangePushA("#fcn#wcoherence#" MW_AT_LOCATION);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&b_gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_x);
  nvtxMarkA("#gpuEmxReset_creal32_T#" MW_AT_LINE);
  gpuEmxReset_creal32_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxReset_real32_T#" MW_AT_LINE);
  gpuEmxReset_real32_T(&gpu_xv);
  nvtxMarkA("#gpuEmxReset_real_T#" MW_AT_LINE);
  gpuEmxReset_real_T(&gpu_scales);
  xposdft_outdatedOnCpu = false;
  xposdft_outdatedOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#emxInitStruct_cwtfilterbank#" MW_AT_LINE);
  emxInitStruct_cwtfilterbank(&fb, &ob_emlrtRTEI, true);
  nvtxMarkA("#b_log2#" MW_AT_LINE);
  b_y = std::trunc(b_log2(static_cast<real_T>(x->size[0])));
  FLimits[0] = varargin_3[0];
  FLimits[1] = varargin_3[1];
  nvtxMarkA("#b_log2#" MW_AT_LINE);
  c_y = std::trunc(b_log2(static_cast<real_T>(x->size[0])));
  if (!(b_y - 1.0 <= c_y - 1.0)) {
    d_y = nullptr;
    m = emlrtCreateCharArray(2, &iv[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &rfmt[0]);
    emlrtAssign(&d_y, m);
    e_y = nullptr;
    m1 = emlrtCreateDoubleScalar(c_y - 1.0);
    emlrtAssign(&e_y, m1);
    nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
    nvtxMarkA("#b_sprintf#" MW_AT_LINE);
    emlrt_marshallIn(b_sprintf(d_y, e_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  guard1 = false;
  if ((varargin_1 == 0.0) || std::isnan(varargin_1)) {
    xposdft_outdatedOnCpu = true;
    k = 0;
    exitg1 = false;
    nvtxRangePushA("#loop#wcoherence_whileloop_1##" MW_AT_LINE);
    while ((!exitg1) && (k < 2)) {
      if (!(varargin_3[k] == 0.0)) {
        xposdft_outdatedOnCpu = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    nvtxRangePop();
    if (xposdft_outdatedOnCpu) {
      nvtxMarkA("#cwtfreqlimits#" MW_AT_LINE);
      wavelet::internal::cwt::cwtfreqlimits(static_cast<real_T>(x->size[0]),
                                            10.0, 1.0, varargin_5, &maxperiod,
                                            &Norig, &ns, &c_y, &minperiod);
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      FLimits[0] = rt_powd_snf(2.0, -(b_y - 1.0)) * c_y;
      FLimits[1] = c_y;
    } else {
      guard1 = true;
    }
  } else {
    guard1 = true;
  }
  if (guard1 && ((!(varargin_1 == 0.0)) && (!std::isnan(varargin_1)))) {
    xposdft_outdatedOnCpu = true;
    k = 0;
    exitg1 = false;
    nvtxRangePushA("#loop#wcoherence_whileloop_0##" MW_AT_LINE);
    while ((!exitg1) && (k < 2)) {
      if (!(varargin_3[k] == 0.0)) {
        xposdft_outdatedOnCpu = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    nvtxRangePop();
    if (xposdft_outdatedOnCpu) {
      nvtxMarkA("#cwtfreqlimits#" MW_AT_LINE);
      wavelet::internal::cwt::cwtfreqlimits(
          static_cast<real_T>(x->size[0]), 10.0, varargin_1, varargin_5,
          &maxperiod, &Norig, &ns, &c_y, &minperiod);
      nvtxMarkA("#rt_powd_snf#" MW_AT_LINE);
      FLimits[0] = rt_powd_snf(2.0, -(b_y - 1.0)) * c_y;
      FLimits[1] = c_y;
    }
  }
  nvtxMarkA("#cwtfilterbank_cwtfilterbank#" MW_AT_LINE);
  cwtfilterbank_cwtfilterbank(&fb, static_cast<real_T>(x->size[0]), FLimits,
                              varargin_1, varargin_5);
  nvtxMarkA("#emxInit_real_T#" MW_AT_LINE);
  emxInit_real_T(&cpu_scales, 2, &fb_emlrtRTEI, true);
  scales_outdatedOnGpu = false;
  i = cpu_scales->size[0] * cpu_scales->size[1];
  cpu_scales->size[0] = 1;
  cpu_scales->size[1] = fb.Scales->size[1];
  nvtxMarkA("#emxEnsureCapacity_real_T#" MW_AT_LINE);
  emxEnsureCapacity_real_T(cpu_scales, i, &fb_emlrtRTEI);
  profileLoopStart("wcoherence_loop_0", __LINE__, (fb.Scales->size[1] - 1) + 1,
                   "");
  for (i = 0; i < fb.Scales->size[1]; i++) {
    cpu_scales->data[i] = fb.Scales->data[i];
    scales_outdatedOnGpu = true;
  }
  profileLoopEnd();
  ns = std::fmin(std::floor(static_cast<real_T>(cpu_scales->size[1]) / 2.0),
                 varargin_5);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cwtx, 2, &pb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&f, 1, &pb_emlrtRTEI, true);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&coitmp, 1, &pb_emlrtRTEI, true);
  nvtxMarkA("#cwtfilterbank_wt#" MW_AT_LINE);
  cwtfilterbank_wt(&fb, x, cwtx, f, coitmp);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&coitmp);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&f);
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&psihat, 2, &gb_emlrtRTEI, true);
  i = psihat->size[0] * psihat->size[1];
  psihat->size[0] = fb.PsiDFT->size[0];
  psihat->size[1] = fb.PsiDFT->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(psihat, i, &gb_emlrtRTEI);
  profileLoopStart("wcoherence_loop_1", __LINE__,
                   (fb.PsiDFT->size[0] * fb.PsiDFT->size[1] - 1) + 1, "");
  for (i = 0; i < fb.PsiDFT->size[0] * fb.PsiDFT->size[1]; i++) {
    psihat->data[i] = static_cast<real32_T>(fb.PsiDFT->data[i]);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_xv, 2, &hb_emlrtRTEI, true);
  xv_outdatedOnCpu = false;
  xv_outdatedOnGpu = false;
  i = cpu_xv->size[0] * cpu_xv->size[1];
  cpu_xv->size[0] = 1;
  cpu_xv->size[1] = y->size[0];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(cpu_xv, i, &hb_emlrtRTEI);
  profileLoopStart("wcoherence_loop_2", __LINE__, (y->size[0] - 1) + 1, "");
  for (i = 0; i < y->size[0]; i++) {
    cpu_xv->data[i] = y->data[i];
    xv_outdatedOnGpu = true;
  }
  profileLoopEnd();
  Norig = fb.SignalLength;
  if (fb.SignalPad > 0.0) {
    boolean_T validLaunchParams;
    c_y = fb.SignalPad;
    if (c_y < 1.0) {
      b_i = 0;
    } else {
      b_i = static_cast<int32_T>(c_y);
    }
    c_y = (static_cast<real_T>(y->size[0]) - fb.SignalPad) + 1.0;
    if (c_y > y->size[0]) {
      k = 0;
      i3 = 1;
      i5 = -1;
    } else {
      k = y->size[0] - 1;
      i3 = -1;
      i5 = static_cast<int32_T>(c_y) - 1;
    }
    nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
    emxInit_real32_T(&cpu_x, 2, &ib_emlrtRTEI, true);
    xposdft_outdatedOnCpu = false;
    i = cpu_x->size[0] * cpu_x->size[1];
    cpu_x->size[0] = 1;
    cpu_x->size[1] = b_i;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_x, i, &ib_emlrtRTEI);
    profileLoopStart("wcoherence_loop_3", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      cpu_x->data[i] = y->data[i];
      xposdft_outdatedOnCpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#computeNumIters#" MW_AT_LINE);
    mwGetLaunchParameters1D(computeNumIters((b_i >> 1) - 1), &grid, &block,
                            2147483647U);
    nvtxMarkA("#gpuEmxEnsureCapacity_real32_T#" MW_AT_LINE);
    gpuEmxEnsureCapacity_real32_T(cpu_x, &gpu_x, !xposdft_outdatedOnCpu);
    if (xposdft_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyCpuToGpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, cpu_x);
    }
    validLaunchParams = mwValidateLaunchParameters(grid, block);
    if (validLaunchParams) {
      nvtxMarkA("#wcoherence_kernel26#" MW_AT_LINE);
      wcoherence_kernel26<<<grid, block>>>(b_i, (b_i >> 1) - 1, gpu_x);
    }
    xposdft_outdatedOnCpu = true;
    xv_outdatedOnGpu = false;
    i = cpu_xv->size[0] * cpu_xv->size[1];
    cpu_xv->size[0] = 1;
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    cpu_xv->size[1] = ((y->size[0] + cpu_x->size[1]) + div_s32(i5 - k, i3)) + 1;
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(cpu_xv, i, &lb_emlrtRTEI);
    b_i = cpu_x->size[1];
    profileLoopStart("wcoherence_loop_7", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      if (xposdft_outdatedOnCpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_x, &gpu_x);
      }
      xposdft_outdatedOnCpu = false;
      cpu_xv->data[i] = cpu_x->data[i];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    b_i = y->size[0];
    profileLoopStart("wcoherence_loop_9", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      cpu_xv->data[i + cpu_x->size[1]] = y->data[i];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#div_s32#" MW_AT_LINE);
    b_i = div_s32(i5 - k, i3);
    profileLoopStart("wcoherence_loop_10", __LINE__, b_i + 1, "");
    for (i = 0; i <= b_i; i++) {
      cpu_xv->data[(i + y->size[0]) + cpu_x->size[1]] = y->data[k + i3 * i];
      xv_outdatedOnGpu = true;
    }
    profileLoopEnd();
    nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
    emxFree_real32_T(&cpu_x);
  }
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cpu_xposdft, 2, &qb_emlrtRTEI, true);
  nvtxMarkA("#fft#" MW_AT_LINE);
  fft(cpu_xv, &xv_outdatedOnCpu, &gpu_xv, &xv_outdatedOnGpu, cpu_xposdft,
      &xposdft_outdatedOnCpu, &gpu_xposdft, &xposdft_outdatedOnGpu);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_xv);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&r, 2, &vb_emlrtRTEI, true);
  if (xposdft_outdatedOnCpu) {
    nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
    gpuEmxMemcpyGpuToCpu_creal32_T(cpu_xposdft, &gpu_xposdft);
  }
  nvtxMarkA("#bsxfun#" MW_AT_LINE);
  bsxfun(cpu_xposdft, psihat, r);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cpu_xposdft);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&psihat);
  nvtxMarkA("#emxInit_creal32_T#" MW_AT_LINE);
  emxInit_creal32_T(&cfspos, 2, &rb_emlrtRTEI, true);
  nvtxMarkA("#ifft#" MW_AT_LINE);
  ifft(r, cfspos);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&r);
  if (fb.SignalPad > 0.0) {
    maxperiod = fb.SignalPad + 1.0;
    c_y = fb.SignalPad + Norig;
    if (maxperiod > c_y) {
      nd2 = 0;
      k = 0;
    } else {
      nd2 = static_cast<int32_T>(maxperiod) - 1;
      k = static_cast<int32_T>(c_y);
    }
    i3 = cfspos->size[0];
    b_i = k - nd2;
    profileLoopStart("wcoherence_loop_4", __LINE__, (b_i - 1) + 1, "");
    for (i = 0; i < b_i; i++) {
      profileLoopStart("wcoherence_loop_6", __LINE__, (i3 - 1) + 1, "");
      for (i5 = 0; i5 < i3; i5++) {
        cfspos->data[i5 + i3 * i] =
            cfspos->data[i5 + cfspos->size[0] * (nd2 + i)];
      }
      profileLoopEnd();
    }
    profileLoopEnd();
    i = cfspos->size[0] * cfspos->size[1];
    cfspos->size[1] = k - nd2;
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cfspos, i, &jb_emlrtRTEI);
  }
  nd2 = static_cast<int32_T>(
      std::trunc(static_cast<real_T>(cwtx->size[0]) / 2.0));
  if (!(ns <= nd2)) {
    f_y = nullptr;
    m2 = emlrtCreateCharArray(2, &iv1[0]);
    emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m2, &rfmt[0]);
    emlrtAssign(&f_y, m2);
    h_y = nullptr;
    m3 = emlrtCreateDoubleScalar(static_cast<real_T>(nd2));
    emlrtAssign(&h_y, m3);
    nvtxMarkA("#emlrt_marshallIn#" MW_AT_LINE);
    nvtxMarkA("#b_sprintf#" MW_AT_LINE);
    emlrt_marshallIn(b_sprintf(f_y, h_y, &emlrtMCI), "<output of sprintf>",
                     unusedExpr);
  }
  nd2 = cwtx->size[0] * cwtx->size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&g_y, 2, &sb_emlrtRTEI, true);
  i = g_y->size[0] * g_y->size[1];
  g_y->size[0] = cwtx->size[0];
  g_y->size[1] = cwtx->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(g_y, i, &h_emlrtRTEI);
  profileLoopStart("wcoherence_loop_5", __LINE__, (nd2 - 1) + 1, "");
  for (k = 0; k < nd2; k++) {
    nvtxMarkA("#rt_hypotf_snf#" MW_AT_LINE);
    g_y->data[k] = rt_hypotf_snf(cwtx->data[k].re, cwtx->data[k].im);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&cpu_y, 2, &sb_emlrtRTEI, true);
  xposdft_outdatedOnGpu = false;
  xposdft_outdatedOnCpu = false;
  i = cpu_y->size[0] * cpu_y->size[1];
  cpu_y->size[0] = g_y->size[0];
  cpu_y->size[1] = g_y->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(cpu_y, i, &kb_emlrtRTEI);
  nd2 = g_y->size[0] * g_y->size[1];
  profileLoopStart("wcoherence_loop_8", __LINE__, (nd2 - 1) + 1, "");
  for (k = 0; k < nd2; k++) {
    cpu_y->data[k] = g_y->data[k] * g_y->data[k];
    xposdft_outdatedOnCpu = true;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&g_y);
  nvtxMarkA("#smoothCFS#" MW_AT_LINE);
  wavelet::internal::cwt::smoothCFS(cpu_y, &xposdft_outdatedOnGpu, &gpu_y,
                                    &xposdft_outdatedOnCpu, cpu_scales,
                                    &gpu_scales, &scales_outdatedOnGpu, ns);
  nd2 = cfspos->size[0] * cfspos->size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&i_y, 2, &tb_emlrtRTEI, true);
  i = i_y->size[0] * i_y->size[1];
  i_y->size[0] = cfspos->size[0];
  i_y->size[1] = cfspos->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(i_y, i, &h_emlrtRTEI);
  profileLoopStart("wcoherence_loop_11", __LINE__, (nd2 - 1) + 1, "");
  for (k = 0; k < nd2; k++) {
    nvtxMarkA("#rt_hypotf_snf#" MW_AT_LINE);
    i_y->data[k] = rt_hypotf_snf(cfspos->data[k].re, cfspos->data[k].im);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&b_cpu_y, 2, &tb_emlrtRTEI, true);
  xv_outdatedOnGpu = false;
  xposdft_outdatedOnCpu = false;
  i = b_cpu_y->size[0] * b_cpu_y->size[1];
  b_cpu_y->size[0] = i_y->size[0];
  b_cpu_y->size[1] = i_y->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(b_cpu_y, i, &kb_emlrtRTEI);
  nd2 = i_y->size[0] * i_y->size[1];
  profileLoopStart("wcoherence_loop_12", __LINE__, (nd2 - 1) + 1, "");
  for (k = 0; k < nd2; k++) {
    b_cpu_y->data[k] = i_y->data[k] * i_y->data[k];
    xposdft_outdatedOnCpu = true;
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&i_y);
  nvtxMarkA("#smoothCFS#" MW_AT_LINE);
  wavelet::internal::cwt::smoothCFS(b_cpu_y, &xv_outdatedOnGpu, &b_gpu_y,
                                    &xposdft_outdatedOnCpu, cpu_scales,
                                    &gpu_scales, &scales_outdatedOnGpu, ns);
  if ((cwtx->size[0] == cfspos->size[0]) &&
      (cwtx->size[1] == cfspos->size[1])) {
    *wcs_outdatedOnCpu = false;
    *wcs_outdatedOnGpu = false;
    i = cpu_wcs->size[0] * cpu_wcs->size[1];
    cpu_wcs->size[0] = cwtx->size[0];
    cpu_wcs->size[1] = cwtx->size[1];
    nvtxMarkA("#emxEnsureCapacity_creal32_T#" MW_AT_LINE);
    emxEnsureCapacity_creal32_T(cpu_wcs, i, &mb_emlrtRTEI);
    profileLoopStart("wcoherence_loop_13", __LINE__,
                     (cwtx->size[0] * cwtx->size[1] - 1) + 1, "");
    for (i = 0; i < cwtx->size[0] * cwtx->size[1]; i++) {
      real32_T cfspos_im;
      real32_T cfspos_re;
      cfspos_re = cfspos->data[i].re;
      cfspos_im = -cfspos->data[i].im;
      cpu_wcs->data[i].re =
          cwtx->data[i].re * cfspos_re - cwtx->data[i].im * cfspos_im;
      cpu_wcs->data[i].im =
          cwtx->data[i].re * cfspos_im + cwtx->data[i].im * cfspos_re;
      *wcs_outdatedOnGpu = true;
    }
    profileLoopEnd();
  } else {
    if (*wcs_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_creal32_T(cpu_wcs, gpu_wcs);
    }
    nvtxMarkA("#binary_expand_op_1#" MW_AT_LINE);
    binary_expand_op_1(cpu_wcs, cwtx, cfspos);
    *wcs_outdatedOnCpu = false;
    *wcs_outdatedOnGpu = true;
  }
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cfspos);
  nvtxMarkA("#emxFree_creal32_T#" MW_AT_LINE);
  emxFree_creal32_T(&cwtx);
  nvtxMarkA("#b_smoothCFS#" MW_AT_LINE);
  wavelet::internal::cwt::b_smoothCFS(cpu_wcs, wcs_outdatedOnCpu, gpu_wcs,
                                      wcs_outdatedOnGpu, cpu_scales,
                                      &gpu_scales, &scales_outdatedOnGpu, ns);
  nvtxMarkA("#emxFree_real_T#" MW_AT_LINE);
  emxFree_real_T(&cpu_scales);
  nd2 = cpu_wcs->size[0] * cpu_wcs->size[1];
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&j_y, 2, &ub_emlrtRTEI, true);
  i = j_y->size[0] * j_y->size[1];
  j_y->size[0] = cpu_wcs->size[0];
  j_y->size[1] = cpu_wcs->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(j_y, i, &h_emlrtRTEI);
  profileLoopStart("wcoherence_loop_14", __LINE__, (nd2 - 1) + 1, "");
  for (k = 0; k < nd2; k++) {
    if (*wcs_outdatedOnCpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_creal32_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_creal32_T(cpu_wcs, gpu_wcs);
    }
    *wcs_outdatedOnCpu = false;
    nvtxMarkA("#rt_hypotf_snf#" MW_AT_LINE);
    j_y->data[k] = rt_hypotf_snf(cpu_wcs->data[k].re, cpu_wcs->data[k].im);
  }
  profileLoopEnd();
  nvtxMarkA("#emxInit_real32_T#" MW_AT_LINE);
  emxInit_real32_T(&k_y, 2, &ub_emlrtRTEI, true);
  i = k_y->size[0] * k_y->size[1];
  k_y->size[0] = j_y->size[0];
  k_y->size[1] = j_y->size[1];
  nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
  emxEnsureCapacity_real32_T(k_y, i, &kb_emlrtRTEI);
  nd2 = j_y->size[0] * j_y->size[1];
  profileLoopStart("wcoherence_loop_15", __LINE__, (nd2 - 1) + 1, "");
  for (k = 0; k < nd2; k++) {
    k_y->data[k] = j_y->data[k] * j_y->data[k];
  }
  profileLoopEnd();
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&j_y);
  if (cpu_y->size[0] == 1) {
    i3 = b_cpu_y->size[0];
  } else {
    i3 = cpu_y->size[0];
  }
  if (cpu_y->size[1] == 1) {
    nd2 = b_cpu_y->size[1];
  } else {
    nd2 = cpu_y->size[1];
  }
  if ((cpu_y->size[0] == b_cpu_y->size[0]) &&
      (cpu_y->size[1] == b_cpu_y->size[1]) && (k_y->size[0] == i3) &&
      (k_y->size[1] == nd2)) {
    i = wcoh->size[0] * wcoh->size[1];
    wcoh->size[0] = k_y->size[0];
    wcoh->size[1] = k_y->size[1];
    nvtxMarkA("#emxEnsureCapacity_real32_T#" MW_AT_LINE);
    emxEnsureCapacity_real32_T(wcoh, i, &nb_emlrtRTEI);
    profileLoopStart("wcoherence_loop_16", __LINE__,
                     (k_y->size[0] * k_y->size[1] - 1) + 1, "");
    for (i = 0; i < k_y->size[0] * k_y->size[1]; i++) {
      if (xposdft_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &gpu_y);
      }
      xposdft_outdatedOnGpu = false;
      if (xv_outdatedOnGpu) {
        nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
        gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_y, &b_gpu_y);
      }
      xv_outdatedOnGpu = false;
      wcoh->data[i] = k_y->data[i] / (cpu_y->data[i] * b_cpu_y->data[i]);
    }
    profileLoopEnd();
  } else {
    if (xposdft_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real32_T(cpu_y, &gpu_y);
    }
    if (xv_outdatedOnGpu) {
      nvtxMarkA("#gpuEmxMemcpyGpuToCpu_real32_T#" MW_AT_LINE);
      gpuEmxMemcpyGpuToCpu_real32_T(b_cpu_y, &b_gpu_y);
    }
    nvtxMarkA("#binary_expand_op#" MW_AT_LINE);
    binary_expand_op(wcoh, k_y, cpu_y, b_cpu_y);
  }
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&k_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&b_cpu_y);
  nvtxMarkA("#emxFree_real32_T#" MW_AT_LINE);
  emxFree_real32_T(&cpu_y);
  nd2 = wcoh->size[0] * wcoh->size[1] - 1;
  profileLoopStart("wcoherence_loop_17", __LINE__, nd2 + 1, "");
  for (b_i = 0; b_i <= nd2; b_i++) {
    if (wcoh->data[b_i] > 1.0F) {
      wcoh->data[b_i] = 1.0F;
    }
  }
  profileLoopEnd();
  nvtxMarkA("#emxFreeStruct_cwtfilterbank#" MW_AT_LINE);
  emxFreeStruct_cwtfilterbank(&fb);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  nvtxMarkA("#gpuEmxFree_real_T#" MW_AT_LINE);
  gpuEmxFree_real_T(&gpu_scales);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_xv);
  nvtxMarkA("#gpuEmxFree_creal32_T#" MW_AT_LINE);
  gpuEmxFree_creal32_T(&gpu_xposdft);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_x);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&gpu_y);
  nvtxMarkA("#gpuEmxFree_real32_T#" MW_AT_LINE);
  gpuEmxFree_real32_T(&b_gpu_y);
  nvtxRangePop();
}

} // namespace coder

// End of code generation (wcoherence.cu)

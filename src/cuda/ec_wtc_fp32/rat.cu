#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// rat.cu
//
// Code generation for function 'rat'
//

// Include files
#include "rat.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
real_T rat(real_T x, real_T *D)
{
  real_T N;
  real_T absx;
  real_T b_x;
  int32_T exponent;
  nvtxRangePushA("#fcn#rat#" MW_AT_LOCATION);
  b_x = x;
  absx = std::abs(x);
  if (std::isinf(absx) || std::isnan(absx)) {
    absx = rtNaN;
  } else if (absx < 4.4501477170144028E-308) {
    absx = 4.94065645841247E-324;
  } else {
    std::frexp(absx, &exponent);
    absx = std::ldexp(1.0, exponent - 53);
  }
  absx = std::fmax(1.0E-12, absx);
  if (std::isinf(x) || std::isnan(x)) {
    N = !std::isnan(x);
    *D = 0.0;
  } else {
    real_T d;
    real_T dlast;
    real_T n;
    real_T nlast;
    n = 1.0;
    d = 0.0;
    nlast = 0.0;
    dlast = 1.0;
    nvtxRangePushA("#loop#rat_whileloop_0##" MW_AT_LINE);
    real_T rndx;
    int32_T exitg1;
    do {
      exitg1 = 0;
      rndx = std::round(b_x);
      if (!std::isinf(b_x)) {
        real_T tmp;
        b_x -= rndx;
        tmp = nlast;
        nlast = n;
        n = n * rndx + tmp;
        tmp = dlast;
        dlast = d;
        d = d * rndx + tmp;
      } else {
        nlast = n;
        dlast = d;
        n = b_x;
        d = 0.0;
      }
      if ((b_x == 0.0) || (std::abs(n / d - x) <= absx)) {
        exitg1 = 1;
      } else {
        b_x = 1.0 / b_x;
      }
    } while (exitg1 == 0);
    nvtxRangePop();
    if (std::isnan(d)) {
      b_x = rtNaN;
    } else if (d < 0.0) {
      b_x = -1.0;
    } else {
      b_x = (d > 0.0);
    }
    N = n / b_x;
    *D = std::abs(d);
  }
  nvtxRangePop();
  return N;
}

} // namespace coder

// End of code generation (rat.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// mod.cu
//
// Code generation for function 'mod'
//

// Include files
#include "mod.h"
#include "rt_nonfinite.h"
#include "MWLocationStringifyNvtx3.h"
#include "nvtx3/nvToolsExt.h"
#include <cmath>

// Function Definitions
//
//
namespace coder {
real_T b_mod(real_T x, real_T y)
{
  real_T r;
  nvtxRangePushA("#fcn#b_mod#" MW_AT_LOCATION);
  r = x;
  if (y == 0.0) {
    if (x == 0.0) {
      r = 0.0;
    }
  } else if (std::isnan(x) || std::isnan(y) || std::isinf(x)) {
    r = rtNaN;
  } else if (x == 0.0) {
    r = 0.0 / y;
  } else if (std::isinf(y)) {
    if (x < 0.0) {
      r = y;
    }
  } else {
    r = std::fmod(x, y);
    if (r == 0.0) {
      r = y * 0.0;
    } else if (x < 0.0) {
      r += y;
    }
  }
  nvtxRangePop();
  return r;
}

} // namespace coder

// End of code generation (mod.cu)

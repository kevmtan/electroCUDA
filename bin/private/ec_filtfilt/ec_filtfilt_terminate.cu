//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_filtfilt_terminate.cu
//
// Code generation for function 'ec_filtfilt_terminate'
//

// Include files
#include "ec_filtfilt_terminate.h"
#include "_coder_ec_filtfilt_mex.h"
#include "ec_filtfilt_data.h"
#include "rt_nonfinite.h"
#include "MWMemoryManager.hpp"

// Function Definitions
void ec_filtfilt_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void ec_filtfilt_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  mwMemoryManagerTerminate();
}

// End of code generation (ec_filtfilt_terminate.cu)

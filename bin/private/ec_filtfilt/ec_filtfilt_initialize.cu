//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_filtfilt_initialize.cu
//
// Code generation for function 'ec_filtfilt_initialize'
//

// Include files
#include "ec_filtfilt_initialize.h"
#include "_coder_ec_filtfilt_mex.h"
#include "ec_filtfilt_data.h"
#include "rt_nonfinite.h"
#include "MWMemoryManager.hpp"

// Function Declarations
static void ec_filtfilt_once();

// Function Definitions
static void ec_filtfilt_once()
{
  mwMemoryManagerInit(256U, 1U, 8U, 2048U);
}

void ec_filtfilt_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "signal_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    ec_filtfilt_once();
  }
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (ec_filtfilt_initialize.cu)

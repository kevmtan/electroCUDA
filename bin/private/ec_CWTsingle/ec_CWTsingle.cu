#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_CWTsingle.cu
//
// Code generation for function 'ec_CWTsingle'
//

// Include files
#include "ec_CWTsingle.h"
#include "ec_CWTsingle_data.h"
#include "ec_CWTsingle_emxutil.h"
#include "ec_CWTsingle_mexutil.h"
#include "ec_CWTsingle_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hipfft/hipfft.h"
#include <cmath>
#include <cstring>

// Variable Definitions
static emlrtMCInfo emlrtMCI{
    53,                                                                // lineNo
    14,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2022b/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtMCInfo b_emlrtMCI{
    55,                                                                // lineNo
    15,                                                                // colNo
    "nonConstPrint",                                                   // fName
    "/usr/local/MATLAB/R2022b/toolbox/eml/lib/matlab/strfun/sprintf.m" // pName
};

static emlrtRTEInfo c_emlrtRTEI{
    1,    // lineNo
    1,    // colNo
    "wt", // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "wt.p" // pName
};

static emlrtRTEInfo d_emlrtRTEI{
    1,        // lineNo
    1,        // colNo
    "realWT", // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "realWT.p" // pName
};

static emlrtRTEInfo e_emlrtRTEI{
    1,             // lineNo
    1,             // colNo
    "cwtFreqGrid", // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "cwtFreqGrid.p" // pName
};

static emlrtRTEInfo f_emlrtRTEI{
    1,                 // lineNo
    1,                 // colNo
    "cuFFTNDCallback", // fName
    "/usr/local/MATLAB/R2022b/toolbox/gpucoder/gpucoder/+gpucoder/+internal/"
    "cuFFTNDCallback.p" // pName
};

static emlrtRTEInfo g_emlrtRTEI{
    1,           // lineNo
    1,           // colNo
    "cwtScales", // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "cwtScales.p" // pName
};

static emlrtRTEInfo h_emlrtRTEI{
    1,              // lineNo
    1,              // colNo
    "freqToScales", // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "freqToScales.p" // pName
};

static emlrtRTEInfo i_emlrtRTEI{
    9,                                                           // lineNo
    5,                                                           // colNo
    "ec_CWTsingle",                                              // fName
    "/home/kt/Gdrive/Git/electroCUDA/bin/src_mex/ec_CWTsingle.m" // pName
};

static emlrtRTEInfo j_emlrtRTEI{
    1,                // lineNo
    1,                // colNo
    "morseBPFilters", // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/wavelet/+wavelet/+internal/+gpu/"
    "morseBPFilters.p" // pName
};

static emlrtRTEInfo k_emlrtRTEI{
    30,                    // lineNo
    21,                    // colNo
    "applyScalarFunction", // fName
    "/usr/local/MATLAB/R2022b/toolbox/eml/eml/+coder/+internal/"
    "applyScalarFunction.m" // pName
};

static emlrtRTEInfo l_emlrtRTEI{
    10,                                                          // lineNo
    1,                                                           // colNo
    "ec_CWTsingle",                                              // fName
    "/home/kt/Gdrive/Git/electroCUDA/bin/src_mex/ec_CWTsingle.m" // pName
};

static emlrtRTEInfo m_emlrtRTEI{
    170,                     // lineNo
    24,                      // colNo
    "combineVectorElements", // fName
    "/usr/local/MATLAB/R2022b/toolbox/eml/lib/matlab/datafun/private/"
    "combineVectorElements.m" // pName
};

static emlrtRTEInfo n_emlrtRTEI{
    97,                      // lineNo
    13,                      // colNo
    "combineVectorElements", // fName
    "/usr/local/MATLAB/R2022b/toolbox/eml/lib/matlab/datafun/private/"
    "combineVectorElements.m" // pName
};

static emlrtRTEInfo o_emlrtRTEI{
    12,                                                          // lineNo
    10,                                                          // colNo
    "ec_CWTsingle",                                              // fName
    "/home/kt/Gdrive/Git/electroCUDA/bin/src_mex/ec_CWTsingle.m" // pName
};

static emlrtRTEInfo p_emlrtRTEI{
    12,                                                          // lineNo
    5,                                                           // colNo
    "ec_CWTsingle",                                              // fName
    "/home/kt/Gdrive/Git/electroCUDA/bin/src_mex/ec_CWTsingle.m" // pName
};

static emlrtRTEInfo q_emlrtRTEI{
    1,                                                           // lineNo
    27,                                                          // colNo
    "ec_CWTsingle",                                              // fName
    "/home/kt/Gdrive/Git/electroCUDA/bin/src_mex/ec_CWTsingle.m" // pName
};

static emlrtRTEInfo r_emlrtRTEI{
    104,                                                 // lineNo
    5,                                                   // colNo
    "cwt",                                               // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/eml/cwt.m" // pName
};

static emlrtRTEInfo s_emlrtRTEI{
    107,                                                 // lineNo
    9,                                                   // colNo
    "cwt",                                               // fName
    "/usr/local/MATLAB/R2022b/toolbox/wavelet/eml/cwt.m" // pName
};

static emlrtRTEInfo t_emlrtRTEI{
    10,                                                          // lineNo
    6,                                                           // colNo
    "ec_CWTsingle",                                              // fName
    "/home/kt/Gdrive/Git/electroCUDA/bin/src_mex/ec_CWTsingle.m" // pName
};

static emlrtRTEInfo u_emlrtRTEI{
    49,                                                              // lineNo
    9,                                                               // colNo
    "mean",                                                          // fName
    "/usr/local/MATLAB/R2022b/toolbox/eml/lib/matlab/datafun/mean.m" // pName
};

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret);

static __global__ void ec_CWTsingle_kernel1(const real_T freqLims[2],
                                            boolean_T b[2]);

static __global__ void ec_CWTsingle_kernel10(const emxArray_real32_T x,
                                             int32_T Norig,
                                             emxArray_real32_T xv);

static __global__ void ec_CWTsingle_kernel11(const emxArray_real32_T x,
                                             const int32_T Norig,
                                             const int32_T Npad, int32_T N,
                                             emxArray_real32_T xv);

static __global__ void ec_CWTsingle_kernel12(const emxArray_real32_T xv,
                                             int32_T b_xv,
                                             emxArray_creal32_T xdft);

static __global__ void ec_CWTsingle_kernel13(const emxArray_creal32_T xdft,
                                             const int32_T Nfilt, int32_T M,
                                             const int32_T cfsdft_dim0,
                                             const int32_T psidft_dim0,
                                             emxArray_real32_T b_psidft,
                                             emxArray_creal32_T cfsdft);

static __global__ void ec_CWTsingle_kernel14(const int32_T cfsdft_dim1,
                                             creal32_T *cfsdft);

static __global__ void ec_CWTsingle_kernel15(const creal32_T *cfsdft,
                                             const emxArray_creal32_T b_cfsdft,
                                             int32_T c_cfsdft,
                                             emxArray_creal32_T cfstmp);

static __global__ void ec_CWTsingle_kernel16(const int32_T cfsdft_dim1,
                                             uint32_T xSize[2]);

static __global__ void ec_CWTsingle_kernel17(int32_T xSize,
                                             emxArray_creal32_T cfstmp);

static __global__ void ec_CWTsingle_kernel18(const emxArray_creal32_T cfstmp,
                                             const int32_T Norig, int32_T Ns,
                                             const int32_T cfs_dim0,
                                             const int32_T cfstmp_dim0,
                                             emxArray_creal32_T cfs);

static __global__ void
ec_CWTsingle_kernel19(const emxArray_creal32_T cfstmp, const int32_T Npad,
                      const int32_T Norig, int32_T Ns, const int32_T cfs_dim0,
                      const int32_T cfstmp_dim0, emxArray_creal32_T cfs);

static __global__ void ec_CWTsingle_kernel2(const real_T maxscale,
                                            real_T freqLims[2]);

static __global__ void
ec_CWTsingle_kernel20(const real32_T maxwavcf, const real_T maxscale,
                      const boolean_T IsOdd, const int32_T M,
                      const int32_T Norig, emxArray_real_T coi);

static __global__ void ec_CWTsingle_kernel21(const emxArray_real_T coi,
                                             int32_T b_coi,
                                             emxArray_real32_T c_coi);

static __global__ void ec_CWTsingle_kernel22(int32_T b_cf,
                                             emxArray_real32_T c_cf,
                                             emxArray_real32_T freqs);

static __global__ void ec_CWTsingle_kernel23(const emxArray_creal32_T cfs,
                                             int32_T N, emxArray_real32_T y);

static __global__ void ec_CWTsingle_kernel24(const emxArray_real32_T y,
                                             const int32_T b_y, int32_T c_y,
                                             const int32_T wt_dim0,
                                             const int32_T y_dim0,
                                             emxArray_real32_T wt);

static __global__ void ec_CWTsingle_kernel25(const emxArray_real32_T wt,
                                             int32_T M, emxArray_real32_T y);

static __global__ void ec_CWTsingle_kernel26(const emxArray_real32_T wt,
                                             const int32_T N, int32_T M,
                                             emxArray_real32_T y);

static __global__ void ec_CWTsingle_kernel27(int32_T xSize,
                                             emxArray_real32_T y);

static __global__ void ec_CWTsingle_kernel28(const real32_T maxwavcf,
                                             const emxArray_real32_T y,
                                             int32_T b_y, emxArray_real32_T wt);

static __global__ void ec_CWTsingle_kernel29(const emxArray_real32_T wt,
                                             int32_T b_wt,
                                             emxArray_real32_T c_wt);

static __global__ void ec_CWTsingle_kernel3(const real_T fs,
                                            real_T freqLims[2]);

static __global__ void ec_CWTsingle_kernel4(const int32_T N,
                                            const real_T maxscale,
                                            const int32_T M,
                                            emxArray_real_T omega);

static __global__ void ec_CWTsingle_kernel5(const real_T freqLims[2],
                                            boolean_T b[2]);

static __global__ void ec_CWTsingle_kernel6(const real_T a0, int32_T Ns,
                                            emxArray_real_T scales);

static __global__ void ec_CWTsingle_kernel7(const real_T fs,
                                            real_T freqLims[2]);

static __global__ void ec_CWTsingle_kernel8(const real_T a0, const real_T s0,
                                            int32_T Ns, emxArray_real_T scales);

static __global__ void
ec_CWTsingle_kernel9(const real_T fs, const emxArray_real_T omega,
                     const emxArray_real_T scales, const int32_T M,
                     const int32_T N, int32_T Ns, const int32_T psidft_dim0,
                     emxArray_real32_T b_cf, emxArray_real32_T b_psidft);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y);

static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y);

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location);

static void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu);

static void gpuEmxEnsureCapacity_real32_T(const emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu);

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu);

static void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu);

static void gpuEmxFree_real32_T(emxArray_real32_T *gpu);

static void gpuEmxFree_real_T(emxArray_real_T *gpu);

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu);

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu);

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu);

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu);

static void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu);

static void gpuEmxReset_real32_T(emxArray_real32_T *gpu);

static void gpuEmxReset_real_T(emxArray_real_T *gpu);

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location);

static real_T rt_powd_snf(real_T u0, real_T u1);

// Function Definitions
static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               emxArray_char_T *ret)
{
  static const int32_T dims[2]{1, -1};
  int32_T iv[2];
  int32_T i;
  const boolean_T bv[2]{false, true};
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "char", false, 2U,
                            (const void *)&dims[0], &bv[0], &iv[0]);
  i = ret->size[0] * ret->size[1];
  ret->size[0] = iv[0];
  ret->size[1] = iv[1];
  emxEnsureCapacity_char_T(ret, i, static_cast<emlrtRTEInfo *>(nullptr));
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret->data[0], 1, false);
  emlrtDestroyArray(&src);
}

static __global__
    __launch_bounds__(32, 1) void ec_CWTsingle_kernel1(const real_T freqLims[2],
                                                       boolean_T b[2])
{
  uint64_T threadId;
  int32_T jj;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  jj = static_cast<int32_T>(threadId);
  if (jj < 2) {
    b[jj] = isnan(freqLims[jj]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel10(
    const emxArray_real32_T x, int32_T Norig, emxArray_real32_T xv)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(Norig - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    xv.data[kk] = x.data[kk];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel11(
    const emxArray_real32_T x, const int32_T Norig, const int32_T Npad,
    int32_T N, emxArray_real32_T xv)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(N);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    if (kk < Npad) {
      xv.data[kk] = x.data[(Npad - kk) - 1];
    } else {
      int32_T b_index;
      boolean_T guard1{false};
      guard1 = false;
      if (kk + 1 > Npad) {
        if (Norig > MAX_int32_T - Npad) {
          b_index = MAX_int32_T;
        } else {
          b_index = Npad + Norig;
        }
        if (kk + 1 <= b_index) {
          b_index = (Npad - kk) - 1;
          if (b_index < 0) {
            b_index = -b_index;
          }
          xv.data[kk] = x.data[b_index - 1];
        } else {
          guard1 = true;
        }
      } else {
        guard1 = true;
      }
      if (guard1) {
        if ((Norig > 0) && (Npad > MAX_int32_T - Norig)) {
          b_index = MAX_int32_T;
        } else {
          b_index = Norig + Npad;
        }
        if (kk + 1 > b_index) {
          if (Norig > 1073741823) {
            b_index = MAX_int32_T;
          } else {
            b_index = Norig << 1;
          }
          b_index = (b_index - kk) - 1;
          if ((b_index < 0) && (Npad < MIN_int32_T - b_index)) {
            b_index = MIN_int32_T;
          } else if ((b_index > 0) && (Npad > MAX_int32_T - b_index)) {
            b_index = MAX_int32_T;
          } else {
            b_index += Npad;
          }
          if (b_index <= 2147483646) {
            b_index++;
          }
          xv.data[kk] = x.data[b_index - 1];
        }
      }
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel12(
    const emxArray_real32_T xv, int32_T b_xv, emxArray_creal32_T xdft)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_xv);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    xdft.data[jj].re = xv.data[jj];
    xdft.data[jj].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel13(
    const emxArray_creal32_T xdft, const int32_T Nfilt, int32_T M,
    const int32_T cfsdft_dim0, const int32_T psidft_dim0,
    emxArray_real32_T b_psidft, emxArray_creal32_T cfsdft)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(Nfilt - 1) + 1UL) *
                (static_cast<uint64_T>(M) + 1UL) -
            1UL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    int32_T kk;
    jj = static_cast<int32_T>(idx % (static_cast<uint64_T>(Nfilt - 1) + 1UL));
    kk = static_cast<int32_T>((idx - static_cast<uint64_T>(jj)) /
                              (static_cast<uint64_T>(Nfilt - 1) + 1UL));
    cfsdft.data[kk + cfsdft_dim0 * jj].re =
        b_psidft.data[kk + psidft_dim0 * jj] * xdft.data[jj].re;
    cfsdft.data[kk + cfsdft_dim0 * jj].im =
        b_psidft.data[kk + psidft_dim0 * jj] * xdft.data[jj].im;
  }
}

static __global__ __launch_bounds__(32, 1) void ec_CWTsingle_kernel14(
    const int32_T cfsdft_dim1, creal32_T *cfsdft)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    cfsdft->re = static_cast<real32_T>(cfsdft_dim1);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel15(
    const creal32_T *cfsdft, const emxArray_creal32_T b_cfsdft,
    int32_T c_cfsdft, emxArray_creal32_T cfstmp)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(c_cfsdft);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    real32_T ai;
    real32_T ar;
    jj = static_cast<int32_T>(idx);
    ar = b_cfsdft.data[jj].re;
    ai = b_cfsdft.data[jj].im;
    if (ai == 0.0F) {
      cfstmp.data[jj].re = ar / cfsdft->re;
      cfstmp.data[jj].im = 0.0F;
    } else if (ar == 0.0F) {
      cfstmp.data[jj].re = 0.0F;
      cfstmp.data[jj].im = ai / cfsdft->re;
    } else {
      cfstmp.data[jj].re = ar / cfsdft->re;
      cfstmp.data[jj].im = ai / cfsdft->re;
    }
  }
}

static __global__ __launch_bounds__(32, 1) void ec_CWTsingle_kernel16(
    const int32_T cfsdft_dim1, uint32_T xSize[2])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    xSize[1] = static_cast<uint32_T>(cfsdft_dim1);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel17(
    int32_T xSize, emxArray_creal32_T cfstmp)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(xSize);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    cfstmp.data[jj].re = 0.0F;
    cfstmp.data[jj].im = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel18(
    const emxArray_creal32_T cfstmp, const int32_T Norig, int32_T Ns,
    const int32_T cfs_dim0, const int32_T cfstmp_dim0, emxArray_creal32_T cfs)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(Norig - 1) + 1UL) *
                (static_cast<uint64_T>(Ns) + 1UL) -
            1UL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_index;
    int32_T jj;
    jj = static_cast<int32_T>(idx % (static_cast<uint64_T>(Norig - 1) + 1UL));
    b_index = static_cast<int32_T>((idx - static_cast<uint64_T>(jj)) /
                                   (static_cast<uint64_T>(Norig - 1) + 1UL));
    cfs.data[b_index + cfs_dim0 * jj] = cfstmp.data[b_index + cfstmp_dim0 * jj];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel19(
    const emxArray_creal32_T cfstmp, const int32_T Npad, const int32_T Norig,
    int32_T Ns, const int32_T cfs_dim0, const int32_T cfstmp_dim0,
    emxArray_creal32_T cfs)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = (static_cast<uint64_T>(Norig - 1) + 1UL) *
                (static_cast<uint64_T>(Ns) + 1UL) -
            1UL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_index;
    int32_T jj;
    jj = static_cast<int32_T>(idx % (static_cast<uint64_T>(Norig - 1) + 1UL));
    b_index = static_cast<int32_T>((idx - static_cast<uint64_T>(jj)) /
                                   (static_cast<uint64_T>(Norig - 1) + 1UL));
    jj = (Npad + jj) + 1;
    cfs.data[b_index + cfs_dim0 * ((jj - Npad) - 1)] =
        cfstmp.data[b_index + cfstmp_dim0 * (jj - 1)];
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_CWTsingle_kernel2(const real_T maxscale,
                                                       real_T freqLims[2])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    freqLims[0] = maxscale;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel20(
    const real32_T maxwavcf, const real_T maxscale, const boolean_T IsOdd,
    const int32_T M, const int32_T Norig, emxArray_real_T coi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(Norig - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    real_T d;
    int32_T b_index;
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    b_index = 0;
    if (kk + 1 <= M) {
      b_index = kk + 1;
    } else if ((kk == M) && (!IsOdd)) {
      b_index = M;
    } else if ((kk + 1 > M) && IsOdd) {
      b_index = Norig - kk;
    } else if ((kk > M) && (!IsOdd)) {
      b_index = Norig - kk;
    }
    d = 1.0 / (0.60690379849267206 * maxscale * static_cast<real_T>(b_index));
    coi.data[kk] = d;
    if (d > maxwavcf) {
      coi.data[kk] = maxwavcf;
    }
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel21(
    const emxArray_real_T coi, int32_T b_coi, emxArray_real32_T c_coi)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_coi);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    c_coi.data[jj] = static_cast<real32_T>(coi.data[jj]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel22(
    int32_T b_cf, emxArray_real32_T c_cf, emxArray_real32_T freqs)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_cf);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    freqs.data[jj] = c_cf.data[jj];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel23(
    const emxArray_creal32_T cfs, int32_T N, emxArray_real32_T y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(N);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_index;
    b_index = static_cast<int32_T>(idx);
    y.data[b_index] = hypotf(cfs.data[b_index].re, cfs.data[b_index].im);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel24(
    const emxArray_real32_T y, const int32_T b_y, int32_T c_y,
    const int32_T wt_dim0, const int32_T y_dim0, emxArray_real32_T wt)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(b_y) + 1UL) * (static_cast<uint64_T>(c_y) + 1UL) -
      1UL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_index;
    int32_T jj;
    b_index = static_cast<int32_T>(idx % (static_cast<uint64_T>(b_y) + 1UL));
    jj = static_cast<int32_T>((idx - static_cast<uint64_T>(b_index)) /
                              (static_cast<uint64_T>(b_y) + 1UL));
    wt.data[b_index + wt_dim0 * jj] = y.data[jj + y_dim0 * b_index];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel25(
    const emxArray_real32_T wt, int32_T M, emxArray_real32_T y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(M);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_index;
    b_index = static_cast<int32_T>(idx);
    y.data[b_index] = wt.data[b_index];
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel26(
    const emxArray_real32_T wt, const int32_T N, int32_T M, emxArray_real32_T y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(M);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_index;
    b_index = static_cast<int32_T>(idx);
    y.data[b_index] += wt.data[N + b_index];
  }
}

static __global__
    __launch_bounds__(1024, 1) void ec_CWTsingle_kernel27(int32_T xSize,
                                                          emxArray_real32_T y)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(xSize);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    y.data[jj] = 0.0F;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel28(
    const real32_T maxwavcf, const emxArray_real32_T y, int32_T b_y,
    emxArray_real32_T wt)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_y);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    wt.data[jj] = y.data[jj] / maxwavcf;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel29(
    const emxArray_real32_T wt, int32_T b_wt, emxArray_real32_T c_wt)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_wt);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    jj = static_cast<int32_T>(idx);
    c_wt.data[jj] = wt.data[jj];
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_CWTsingle_kernel3(const real_T fs,
                                                       real_T freqLims[2])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    freqLims[1] = fs / 2.0;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel4(
    const int32_T N, const real_T maxscale, const int32_T M,
    emxArray_real_T omega)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(M);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T b_index;
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    omega.data[kk] = static_cast<real_T>(kk) * maxscale;
    b_index = kk + M;
    if (b_index > 2147483645) {
      b_index = MAX_int32_T;
    } else {
      b_index += 2;
    }
    if (b_index <= N) {
      b_index = kk + M;
      if (b_index > 2147483645) {
        b_index = MAX_int32_T;
      } else {
        b_index += 2;
      }
      omega.data[b_index - 1] = 0.0;
    }
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_CWTsingle_kernel5(const real_T freqLims[2],
                                                       boolean_T b[2])
{
  uint64_T threadId;
  int32_T jj;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  jj = static_cast<int32_T>(threadId);
  if (jj < 2) {
    b[jj] = isnan(freqLims[jj]);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel6(
    const real_T a0, int32_T Ns, emxArray_real_T scales)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(Ns - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    scales.data[kk] =
        0.68998891650046823 * pow(a0, (static_cast<real_T>(kk) + 1.0) - 1.0);
  }
}

static __global__
    __launch_bounds__(32, 1) void ec_CWTsingle_kernel7(const real_T fs,
                                                       real_T freqLims[2])
{
  uint64_T threadId;
  int32_T jj;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  jj = static_cast<int32_T>(threadId);
  if (jj < 2) {
    freqLims[jj] = freqLims[jj] / fs * 2.0 * 3.1415926535897931;
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel8(
    const real_T a0, const real_T s0, int32_T Ns, emxArray_real_T scales)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(Ns - 1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T kk;
    kk = static_cast<int32_T>(idx);
    scales.data[kk] = s0 * pow(a0, (static_cast<real_T>(kk) + 1.0) - 1.0);
  }
}

static __global__ __launch_bounds__(1024, 1) void ec_CWTsingle_kernel9(
    const real_T fs, const emxArray_real_T omega, const emxArray_real_T scales,
    const int32_T M, const int32_T N, int32_T Ns, const int32_T psidft_dim0,
    emxArray_real32_T b_cf, emxArray_real32_T b_psidft)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd =
      (static_cast<uint64_T>(N - 1) + 1UL) * (static_cast<uint64_T>(Ns) + 1UL) -
      1UL;
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T jj;
    int32_T kk;
    jj = static_cast<int32_T>(idx % (static_cast<uint64_T>(N - 1) + 1UL));
    kk = static_cast<int32_T>((idx - static_cast<uint64_T>(jj)) /
                              (static_cast<uint64_T>(N - 1) + 1UL));
    if (jj + 1 > M) {
      b_psidft.data[kk + psidft_dim0 * jj] = 0.0F;
    } else {
      real_T d;
      real_T d1;
      d = scales.data[kk];
      d1 = omega.data[jj];
      b_psidft.data[kk + psidft_dim0 * jj] = static_cast<real32_T>(
          0.0050536085896138571 * exp(20.0 * log(d * d1) - pow(d * d1, 3.0)));
    }
    if (jj + 1 == 1) {
      b_cf.data[kk] = static_cast<real32_T>(
          1.8820720577620569 / scales.data[kk] / 6.2831853071795862 * fs);
    }
  }
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             emxArray_char_T *y)
{
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *tmpStr, const char_T *identifier,
                             emxArray_char_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(tmpStr), &thisId, y);
  emlrtDestroyArray(&tmpStr);
}

static const mxArray *feval(const mxArray *m1, const mxArray *m2,
                            const mxArray *m3, emlrtMCInfo *location)
{
  const mxArray *pArrays[3];
  const mxArray *m;
  pArrays[0] = m1;
  pArrays[1] = m2;
  pArrays[2] = m3;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 3, &pArrays[0],
                               "feval", true, location);
}

static void gpuEmxEnsureCapacity_creal32_T(const emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu)
{
  creal32_T *newData;
  if (gpu->data == 0) {
    newData = 0UL;
    mwCudaMalloc(&newData, static_cast<uint64_T>(
                               static_cast<uint32_T>(cpu->allocatedSize) *
                               sizeof(creal32_T)));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0UL;
      mwCudaMalloc(&newData, static_cast<uint64_T>(
                                 static_cast<uint32_T>(cpu->allocatedSize) *
                                 sizeof(creal32_T)));
      hipMemcpy(newData, gpu->data,
                 static_cast<uint32_T>(actualSizeGpu) * sizeof(creal32_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        mwCudaFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxEnsureCapacity_real32_T(const emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu)
{
  real32_T *newData;
  if (gpu->data == 0) {
    newData = 0UL;
    mwCudaMalloc(&newData, static_cast<uint64_T>(
                               static_cast<uint32_T>(cpu->allocatedSize) *
                               sizeof(real32_T)));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0UL;
      mwCudaMalloc(&newData, static_cast<uint64_T>(
                                 static_cast<uint32_T>(cpu->allocatedSize) *
                                 sizeof(real32_T)));
      hipMemcpy(newData, gpu->data,
                 static_cast<uint32_T>(actualSizeGpu) * sizeof(real32_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        mwCudaFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxEnsureCapacity_real_T(const emxArray_real_T *cpu,
                                        emxArray_real_T *gpu)
{
  real_T *newData;
  if (gpu->data == 0) {
    newData = 0UL;
    mwCudaMalloc(&newData, static_cast<uint64_T>(
                               static_cast<uint32_T>(cpu->allocatedSize) *
                               sizeof(real_T)));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0UL;
      mwCudaMalloc(&newData, static_cast<uint64_T>(
                                 static_cast<uint32_T>(cpu->allocatedSize) *
                                 sizeof(real_T)));
      hipMemcpy(newData, gpu->data,
                 static_cast<uint32_T>(actualSizeGpu) * sizeof(real_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        mwCudaFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxFree_creal32_T(emxArray_creal32_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    mwCudaFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxFree_real32_T(emxArray_real32_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    mwCudaFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxFree_real_T(emxArray_real_T *gpu)
{
  if (gpu->data != (void *)4207599121UL) {
    mwCudaFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxMemcpyCpuToGpu_creal32_T(emxArray_creal32_T *gpu,
                                           const emxArray_creal32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      mwCudaFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    mwCudaMalloc(&gpu->data, static_cast<uint64_T>(
                                 static_cast<uint32_T>(gpu->allocatedSize) *
                                 sizeof(creal32_T)));
  }
  hipMemcpy(gpu->data, cpu->data,
             static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyCpuToGpu_real32_T(emxArray_real32_T *gpu,
                                          const emxArray_real32_T *cpu)
{
  int32_T actualSize;
  int32_T i;
  if (gpu->numDimensions < cpu->numDimensions) {
    gpu->numDimensions = cpu->numDimensions;
    emlrtFreeMex(gpu->size);
    gpu->size = static_cast<int32_T *>(emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T)));
  } else {
    gpu->numDimensions = cpu->numDimensions;
  }
  actualSize = 1;
  for (i = 0; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
    gpu->size[i] = cpu->size[i];
  }
  if (gpu->allocatedSize < actualSize) {
    if (gpu->canFreeData) {
      mwCudaFree(gpu->data);
    }
    i = cpu->allocatedSize;
    if (i < actualSize) {
      i = actualSize;
    }
    gpu->allocatedSize = i;
    gpu->canFreeData = true;
    mwCudaMalloc(&gpu->data, static_cast<uint64_T>(
                                 static_cast<uint32_T>(gpu->allocatedSize) *
                                 sizeof(real32_T)));
  }
  hipMemcpy(gpu->data, cpu->data,
             static_cast<uint32_T>(actualSize) * sizeof(real32_T),
             hipMemcpyHostToDevice);
}

static void gpuEmxMemcpyGpuToCpu_creal32_T(emxArray_creal32_T *cpu,
                                           emxArray_creal32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data,
             static_cast<uint32_T>(actualSize) * sizeof(creal32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxMemcpyGpuToCpu_real32_T(emxArray_real32_T *cpu,
                                          emxArray_real32_T *gpu)
{
  int32_T actualSize;
  actualSize = 1;
  for (int32_T i{0}; i < cpu->numDimensions; i++) {
    actualSize *= cpu->size[i];
  }
  hipMemcpy(cpu->data, gpu->data,
             static_cast<uint32_T>(actualSize) * sizeof(real32_T),
             hipMemcpyDeviceToHost);
}

static void gpuEmxReset_creal32_T(emxArray_creal32_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_creal32_T));
}

static void gpuEmxReset_real32_T(emxArray_real32_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real32_T));
}

static void gpuEmxReset_real_T(emxArray_real_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_real_T));
}

static const mxArray *length(const mxArray *m1, emlrtMCInfo *location)
{
  const mxArray *m;
  const mxArray *pArray;
  pArray = m1;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 1, &pArray, "length",
                               true, location);
}

static real_T rt_powd_snf(real_T u0, real_T u1)
{
  real_T y;
  if (std::isnan(u0) || std::isnan(u1)) {
    y = rtNaN;
  } else {
    real_T b;
    real_T c;
    b = std::abs(u0);
    c = std::abs(u1);
    if (std::isinf(u1)) {
      if (b == 1.0) {
        y = 1.0;
      } else if (b > 1.0) {
        if (u1 > 0.0) {
          y = rtInf;
        } else {
          y = 0.0;
        }
      } else if (u1 > 0.0) {
        y = 0.0;
      } else {
        y = rtInf;
      }
    } else if (c == 0.0) {
      y = 1.0;
    } else if (c == 1.0) {
      if (u1 > 0.0) {
        y = u0;
      } else {
        y = 1.0 / u0;
      }
    } else if (u1 == 2.0) {
      y = u0 * u0;
    } else if ((u1 == 0.5) && (u0 >= 0.0)) {
      y = std::sqrt(u0);
    } else if ((u0 < 0.0) && (u1 > std::floor(u1))) {
      y = rtNaN;
    } else {
      y = std::pow(u0, u1);
    }
  }
  return y;
}

void ec_CWTsingle(const emxArray_real32_T *x, real_T fs, real_T freqLims[2],
                  boolean_T doAvg, real_T freqsPerOctave, emxArray_real32_T *wt,
                  emxArray_real32_T *freqs, emxArray_real32_T *coi)
{
  static const int32_T iv[2]{1, 7};
  static const int32_T iv1[2]{1, 5};
  static const char_T u[7]{'s', 'p', 'r', 'i', 'n', 't', 'f'};
  static const char_T formatSpec[5]{'%', '2', '.', '2', 'f'};
  dim3 block;
  dim3 grid;
  emxArray_char_T *b_tmpStr;
  emxArray_creal32_T gpu_cfs;
  emxArray_creal32_T gpu_cfsdft;
  emxArray_creal32_T gpu_cfstmp;
  emxArray_creal32_T gpu_xdft;
  emxArray_creal32_T *cfs;
  emxArray_creal32_T *cfsdft;
  emxArray_creal32_T *cfstmp;
  emxArray_creal32_T *xdft;
  emxArray_real32_T b_gpu_coi;
  emxArray_real32_T b_gpu_wt;
  emxArray_real32_T b_gpu_y;
  emxArray_real32_T gpu_cf;
  emxArray_real32_T gpu_freqs;
  emxArray_real32_T gpu_psidft;
  emxArray_real32_T gpu_wt;
  emxArray_real32_T gpu_x;
  emxArray_real32_T gpu_xv;
  emxArray_real32_T gpu_y;
  emxArray_real32_T *b_wt;
  emxArray_real32_T *d_y;
  emxArray_real32_T *e_y;
  emxArray_real32_T *xv;
  emxArray_real_T gpu_coi;
  emxArray_real_T gpu_omega;
  emxArray_real_T gpu_scales;
  emxArray_real_T *b_coi;
  emxArray_real_T *omega;
  emxArray_real_T *scales;
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *m;
  const mxArray *tmpStr;
  const mxArray *y;
  creal32_T *b_gpu_cfsdft;
  real_T(*gpu_freqLims)[2];
  int32_T M;
  int32_T N;
  int32_T Nfilt;
  int32_T Norig;
  int32_T Npad;
  int32_T Ns;
  int32_T k;
  real32_T maxwavcf;
  uint32_T xSize[2];
  uint32_T(*gpu_xSize)[2];
  boolean_T b[2];
  boolean_T(*gpu_b)[2];
  boolean_T IsOdd;
  boolean_T b_dirtyOnGpu;
  boolean_T coi_dirtyOnGpu;
  boolean_T freqs_dirtyOnGpu;
  boolean_T wt_dirtyOnCpu;
  boolean_T wt_dirtyOnGpu;
  boolean_T xSize_dirtyOnCpu;
  boolean_T xdft_dirtyOnCpu;
  gpuEmxReset_real32_T(&b_gpu_wt);
  gpuEmxReset_real32_T(&b_gpu_y);
  gpuEmxReset_real32_T(&gpu_wt);
  gpuEmxReset_real32_T(&gpu_y);
  gpuEmxReset_real32_T(&gpu_freqs);
  gpuEmxReset_real32_T(&b_gpu_coi);
  gpuEmxReset_real_T(&gpu_coi);
  gpuEmxReset_real32_T(&gpu_cf);
  gpuEmxReset_creal32_T(&gpu_cfs);
  gpuEmxReset_real_T(&gpu_scales);
  gpuEmxReset_creal32_T(&gpu_cfstmp);
  mwCudaMalloc(&b_gpu_cfsdft, 8UL);
  mwCudaMalloc(&gpu_xSize, 8UL);
  gpuEmxReset_creal32_T(&gpu_cfsdft);
  gpuEmxReset_real32_T(&gpu_psidft);
  gpuEmxReset_creal32_T(&gpu_xdft);
  gpuEmxReset_real_T(&gpu_omega);
  gpuEmxReset_real32_T(&gpu_xv);
  gpuEmxReset_real32_T(&gpu_x);
  mwCudaMalloc(&gpu_b, 2UL);
  mwCudaMalloc(&gpu_freqLims, 16UL);
  xSize_dirtyOnCpu = false;
  xdft_dirtyOnCpu = false;
  wt_dirtyOnGpu = false;
  wt_dirtyOnCpu = true;
  freqs_dirtyOnGpu = false;
  coi_dirtyOnGpu = false;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  //  Input validation
  if ((!(fs != 0.0)) || (fs < 1.0)) {
    fs = 1000.0;
  }
  if (freqLims[0] <= 0.0) {
    freqLims[0] = 1.0;
  }
  if (freqLims[1] > fs / 2.0) {
    freqLims[1] = fs / 2.0;
  }
  //  Initialize
  //  Add kernelfun pragma to trigger kernel creation
  if (!psidft_not_empty) {
    real_T maxscale;
    real_T s0;
    boolean_T exitg1;
    N = static_cast<int32_T>(std::trunc(static_cast<real_T>(x->size[0]) / 2.0));
    hipMemcpy(*gpu_freqLims, freqLims, 16UL, hipMemcpyHostToDevice);
    ec_CWTsingle_kernel1<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_freqLims,
                                                                  *gpu_b);
    b_dirtyOnGpu = true;
    IsOdd = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k <= 1)) {
      if (b_dirtyOnGpu) {
        hipMemcpy(b, *gpu_b, 2UL, hipMemcpyDeviceToHost);
      }
      b_dirtyOnGpu = false;
      if (!b[k]) {
        IsOdd = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    if (!IsOdd) {
      maxscale = static_cast<real_T>(x->size[0]) / 11.001547073884861;
      if (maxscale <
          0.68998891650046823 * rt_powd_snf(2.0, 1.0 / freqsPerOctave)) {
        maxscale = 0.68998891650046823 * rt_powd_snf(2.0, 1.0 / freqsPerOctave);
      }
      maxscale = 1.0 / (maxscale * 3.3384403542183319) * fs;
      if (freqLims[0] < maxscale) {
        ec_CWTsingle_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            maxscale, *gpu_freqLims);
        hipMemcpy(freqLims, *gpu_freqLims, 16UL, hipMemcpyDeviceToHost);
      }
      if (freqLims[1] > fs / 2.0) {
        ec_CWTsingle_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
            fs, *gpu_freqLims);
        hipMemcpy(freqLims, *gpu_freqLims, 16UL, hipMemcpyDeviceToHost);
      }
      if (freqLims[1] == 0.0) {
        s0 = rtMinusInf;
      } else if (freqLims[1] < 0.0) {
        s0 = rtNaN;
      } else if ((!std::isinf(freqLims[1])) && (!std::isnan(freqLims[1]))) {
        maxscale = std::frexp(freqLims[1], &M);
        if (maxscale == 0.5) {
          s0 = static_cast<real_T>(M) - 1.0;
        } else if ((M == 1) && (maxscale < 0.75)) {
          s0 = std::log(2.0 * maxscale) / 0.69314718055994529;
        } else {
          s0 =
              std::log(maxscale) / 0.69314718055994529 + static_cast<real_T>(M);
        }
      } else {
        s0 = freqLims[1];
      }
      if ((!std::isinf(freqLims[0])) && (!std::isnan(freqLims[0]))) {
        maxscale = std::frexp(freqLims[0], &Nfilt);
        if (maxscale == 0.5) {
          maxscale = static_cast<real_T>(Nfilt) - 1.0;
        } else if ((Nfilt == 1) && (maxscale < 0.75)) {
          maxscale = std::log(2.0 * maxscale) / 0.69314718055994529;
        } else {
          maxscale = std::log(maxscale) / 0.69314718055994529 +
                     static_cast<real_T>(Nfilt);
        }
      } else {
        maxscale = freqLims[0];
      }
      if (!(s0 - maxscale >= 1.0 / freqsPerOctave)) {
        tmpStr = nullptr;
        y = nullptr;
        m = emlrtCreateCharArray(2, &iv[0]);
        emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 7, m, &u[0]);
        emlrtAssign(&y, m);
        b_y = nullptr;
        m = emlrtCreateCharArray(2, &iv1[0]);
        emlrtInitCharArrayR2013a(emlrtRootTLSGlobal, 5, m, &formatSpec[0]);
        emlrtAssign(&b_y, m);
        c_y = nullptr;
        m = emlrtCreateDoubleScalar(1.0 / freqsPerOctave);
        emlrtAssign(&c_y, m);
        emlrtAssign(&tmpStr, feval(y, b_y, c_y, &emlrtMCI));
        emlrt_marshallIn(length(emlrtAlias(tmpStr), &b_emlrtMCI),
                         "<output of length>");
        emxInit_char_T(&b_tmpStr, 2, &q_emlrtRTEI, true);
        emlrt_marshallIn(emlrtAlias(tmpStr), "tmpStr", b_tmpStr);
        emxFree_char_T(&b_tmpStr);
        emlrtDestroyArray(&tmpStr);
      }
    }
    if (x->size[0] > 100000) {
      maxscale = std::frexp(static_cast<real_T>(x->size[0]), &Npad);
      if (maxscale == 0.5) {
        maxscale = static_cast<real_T>(Npad) - 1.0;
      } else {
        maxscale = std::log(maxscale) / 0.69314718055994529 +
                   static_cast<real_T>(Npad);
      }
      maxscale = std::ceil(maxscale);
      N = static_cast<int32_T>(maxscale);
    }
    N <<= 1;
    if ((x->size[0] > 0) && (N > MAX_int32_T - x->size[0])) {
      N = MAX_int32_T;
    } else {
      N += x->size[0];
    }
    M = static_cast<int32_T>(std::trunc(static_cast<real_T>(N) / 2.0));
    emxInit_real_T(&omega, 2, &r_emlrtRTEI, true);
    k = omega->size[0] * omega->size[1];
    omega->size[0] = 1;
    omega->size[1] = N;
    emxEnsureCapacity_real_T(omega, k, &e_emlrtRTEI);
    gpuEmxEnsureCapacity_real_T(omega, &gpu_omega);
    b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(M + 1L), &grid,
                                           &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      ec_CWTsingle_kernel4<<<grid, block>>>(
          N, 6.2831853071795862 / static_cast<real_T>(N), M, gpu_omega);
    }
    ec_CWTsingle_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_freqLims,
                                                                  *gpu_b);
    b_dirtyOnGpu = true;
    IsOdd = true;
    k = 0;
    exitg1 = false;
    while ((!exitg1) && (k <= 1)) {
      if (b_dirtyOnGpu) {
        hipMemcpy(b, *gpu_b, 2UL, hipMemcpyDeviceToHost);
      }
      b_dirtyOnGpu = false;
      if (!b[k]) {
        IsOdd = false;
        exitg1 = true;
      } else {
        k++;
      }
    }
    emxInit_real_T(&scales, 2, &s_emlrtRTEI, true);
    if (!IsOdd) {
      ec_CWTsingle_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          fs, *gpu_freqLims);
      hipMemcpy(freqLims, *gpu_freqLims, 16UL, hipMemcpyDeviceToHost);
      s0 = 1.8820720577620569 / freqLims[1];
      maxscale = 1.8820720577620569 / freqLims[0] / s0;
      if (maxscale == 0.0) {
        maxscale = rtMinusInf;
      } else if (maxscale < 0.0) {
        maxscale = rtNaN;
      } else if ((!std::isinf(maxscale)) && (!std::isnan(maxscale))) {
        maxscale = std::frexp(maxscale, &Ns);
        if (maxscale == 0.5) {
          maxscale = static_cast<real_T>(Ns) - 1.0;
        } else if ((Ns == 1) && (maxscale < 0.75)) {
          maxscale = std::log(2.0 * maxscale) / 0.69314718055994529;
        } else {
          maxscale = std::log(maxscale) / 0.69314718055994529 +
                     static_cast<real_T>(Ns);
        }
      }
      maxscale = std::floor(freqsPerOctave * maxscale);
      if (maxscale < 2.147483648E+9) {
        if (maxscale >= -2.147483648E+9) {
          N = static_cast<int32_T>(maxscale);
        } else {
          N = MIN_int32_T;
        }
      } else if (maxscale >= 2.147483648E+9) {
        N = MAX_int32_T;
      } else {
        N = 0;
      }
      if (N > 2147483646) {
        Ns = MAX_int32_T;
      } else {
        Ns = N + 1;
      }
      k = scales->size[0] * scales->size[1];
      scales->size[0] = 1;
      scales->size[1] = Ns;
      emxEnsureCapacity_real_T(scales, k, &h_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(scales, &gpu_scales);
      b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>((Ns - 1) + 1L),
                                             &grid, &block, 1024U, 65535U);
      if (b_dirtyOnGpu) {
        ec_CWTsingle_kernel8<<<grid, block>>>(
            rt_powd_snf(2.0, 1.0 / freqsPerOctave), s0, Ns, gpu_scales);
      }
    } else {
      maxscale = static_cast<real_T>(x->size[0]) / 11.001547073884861;
      if (maxscale <
          0.68998891650046823 * rt_powd_snf(2.0, 1.0 / freqsPerOctave)) {
        maxscale = 0.68998891650046823 * rt_powd_snf(2.0, 1.0 / freqsPerOctave);
      }
      maxscale /= 0.68998891650046823;
      if (maxscale == 0.0) {
        maxscale = rtMinusInf;
      } else if ((!std::isinf(maxscale)) && (!std::isnan(maxscale))) {
        maxscale = std::frexp(maxscale, &Ns);
        if (maxscale == 0.5) {
          maxscale = static_cast<real_T>(Ns) - 1.0;
        } else if ((Ns == 1) && (maxscale < 0.75)) {
          maxscale = std::log(2.0 * maxscale) / 0.69314718055994529;
        } else {
          maxscale = std::log(maxscale) / 0.69314718055994529 +
                     static_cast<real_T>(Ns);
        }
      }
      maxscale = std::floor(std::fmax(maxscale, 1.0 / freqsPerOctave) *
                            freqsPerOctave);
      if (maxscale < 2.147483648E+9) {
        if (maxscale >= -2.147483648E+9) {
          N = static_cast<int32_T>(maxscale);
        } else {
          N = MIN_int32_T;
        }
      } else if (maxscale >= 2.147483648E+9) {
        N = MAX_int32_T;
      } else {
        N = 0;
      }
      if (N > 2147483646) {
        Ns = MAX_int32_T;
      } else {
        Ns = N + 1;
      }
      k = scales->size[0] * scales->size[1];
      scales->size[0] = 1;
      scales->size[1] = Ns;
      emxEnsureCapacity_real_T(scales, k, &g_emlrtRTEI);
      gpuEmxEnsureCapacity_real_T(scales, &gpu_scales);
      b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>((Ns - 1) + 1L),
                                             &grid, &block, 1024U, 65535U);
      if (b_dirtyOnGpu) {
        ec_CWTsingle_kernel6<<<grid, block>>>(
            rt_powd_snf(2.0, 1.0 / freqsPerOctave), Ns, gpu_scales);
      }
    }
    Ns = scales->size[1] - 1;
    N = omega->size[1];
    k = psidft->size[0] * psidft->size[1];
    psidft->size[0] = scales->size[1];
    psidft->size[1] = omega->size[1];
    emxEnsureCapacity_real32_T(psidft, k, &j_emlrtRTEI);
    k = cf->size[0];
    cf->size[0] = scales->size[1];
    emxEnsureCapacity_real32_T(cf, k, &j_emlrtRTEI);
    M = static_cast<int32_T>(
            std::trunc(static_cast<real_T>(omega->size[1]) / 2.0)) +
        1;
    k = psidft->size[0];
    b_dirtyOnGpu =
        mwGetLaunchParameters1D(static_cast<real_T>(((N - 1) + 1L) * (Ns + 1L)),
                                &grid, &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_psidft, psidft);
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_cf, cf);
      ec_CWTsingle_kernel9<<<grid, block>>>(fs, gpu_omega, gpu_scales, M, N, Ns,
                                            k, gpu_cf, gpu_psidft);
      gpuEmxMemcpyGpuToCpu_real32_T(psidft, &gpu_psidft);
      gpuEmxMemcpyGpuToCpu_real32_T(cf, &gpu_cf);
    }
    emxFree_real_T(&scales);
    emxFree_real_T(&omega);
    psidft_not_empty = ((psidft->size[0] != 0) && (psidft->size[1] != 0));
  }
  Norig = x->size[0];
  Npad = static_cast<int32_T>(
      std::trunc(static_cast<real_T>(psidft->size[1] - x->size[0]) / 2.0));
  Ns = psidft->size[0] - 1;
  emxInit_real32_T(&xv, 2, &c_emlrtRTEI, true);
  k = xv->size[0] * xv->size[1];
  xv->size[0] = 1;
  xv->size[1] = psidft->size[1];
  emxEnsureCapacity_real32_T(xv, k, &c_emlrtRTEI);
  gpuEmxEnsureCapacity_real32_T(xv, &gpu_xv);
  emxInit_creal32_T(&cfs, 2, &q_emlrtRTEI, true);
  k = cfs->size[0] * cfs->size[1];
  cfs->size[0] = psidft->size[0];
  cfs->size[1] = x->size[0];
  emxEnsureCapacity_creal32_T(cfs, k, &c_emlrtRTEI);
  gpuEmxEnsureCapacity_creal32_T(cfs, &gpu_cfs);
  emxInit_creal32_T(&cfstmp, 2, &c_emlrtRTEI, true);
  N = xv->size[1] - 1;
  if (Npad > 0) {
    b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                           &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, x);
      ec_CWTsingle_kernel11<<<grid, block>>>(gpu_x, Norig, Npad, N, gpu_xv);
    }
  } else {
    b_dirtyOnGpu = mwGetLaunchParameters1D(
        static_cast<real_T>((Norig - 1) + 1L), &grid, &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      gpuEmxMemcpyCpuToGpu_real32_T(&gpu_x, x);
      ec_CWTsingle_kernel10<<<grid, block>>>(gpu_x, Norig, gpu_xv);
    }
  }
  emxInit_creal32_T(&xdft, 2, &d_emlrtRTEI, true);
  if (xv->size[1] == 0) {
    xdft->size[0] = 1;
    xdft->size[1] = 0;
  } else if (xv->size[1] -
                 (static_cast<int32_T>(static_cast<uint32_T>(xv->size[1]) >> 1)
                  << 1) ==
             1) {
    hipfftHandle fftPlanHandle;
    k = xdft->size[0] * xdft->size[1];
    xdft->size[0] = 1;
    xdft->size[1] = xv->size[1];
    emxEnsureCapacity_creal32_T(xdft, k, &d_emlrtRTEI);
    gpuEmxEnsureCapacity_creal32_T(xdft, &gpu_xdft);
    N = xv->size[1] - 1;
    b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                           &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      ec_CWTsingle_kernel12<<<grid, block>>>(gpu_xv, N, gpu_xdft);
    }
    N = xdft->size[1];
    fftPlanHandle = acquireCUFFTPlan(1, &N, &N, 1, 1, HIPFFT_C2C, 1);
    hipfftExecC2C(fftPlanHandle, (hipfftComplex *)&gpu_xdft.data[0],
                 (hipfftComplex *)&gpu_xdft.data[0], HIPFFT_FORWARD);
  } else {
    N = xv->size[1];
    for (Nfilt = 0; Nfilt < 2; Nfilt++) {
      xSize[Nfilt] = static_cast<uint32_T>(xv->size[Nfilt]);
      xSize_dirtyOnCpu = true;
    }
    hipfftHandle b_fftPlanHandle;
    k = xdft->size[0] * xdft->size[1];
    xdft->size[0] = 1;
    xdft->size[1] = static_cast<int32_T>(xSize[1]);
    emxEnsureCapacity_creal32_T(xdft, k, &f_emlrtRTEI);
    gpuEmxEnsureCapacity_creal32_T(xdft, &gpu_xdft);
    b_fftPlanHandle = acquireCUFFTPlan(1, &N, &N, 1, 1, HIPFFT_R2C, 1);
    hipfftExecR2C(b_fftPlanHandle, (hipfftReal *)&gpu_xv.data[0],
                 (hipfftComplex *)&gpu_xdft.data[0]);
    b_dirtyOnGpu = true;
    N = static_cast<int32_T>(
        static_cast<real_T>(static_cast<int32_T>(xSize[1]) + 1) / 2.0);
    for (Nfilt = 0; Nfilt <= N - 2; Nfilt++) {
      if (b_dirtyOnGpu) {
        gpuEmxMemcpyGpuToCpu_creal32_T(xdft, &gpu_xdft);
      }
      xdft->data[(static_cast<int32_T>(xSize[1]) - Nfilt) - 1].re =
          xdft->data[Nfilt + 1].re;
      xdft->data[(static_cast<int32_T>(xSize[1]) - Nfilt) - 1].im =
          -xdft->data[Nfilt + 1].im;
      b_dirtyOnGpu = false;
      xdft_dirtyOnCpu = true;
    }
  }
  emxFree_real32_T(&xv);
  M = psidft->size[0] - 1;
  Nfilt = psidft->size[1];
  emxInit_creal32_T(&cfsdft, 2, &d_emlrtRTEI, true);
  k = cfsdft->size[0] * cfsdft->size[1];
  cfsdft->size[0] = psidft->size[0];
  cfsdft->size[1] = psidft->size[1];
  emxEnsureCapacity_creal32_T(cfsdft, k, &d_emlrtRTEI);
  gpuEmxEnsureCapacity_creal32_T(cfsdft, &gpu_cfsdft);
  N = cfsdft->size[0];
  k = psidft->size[0];
  b_dirtyOnGpu = mwGetLaunchParameters1D(
      static_cast<real_T>(((Nfilt - 1) + 1L) * (M + 1L)), &grid, &block, 1024U,
      65535U);
  if (b_dirtyOnGpu) {
    if (xdft_dirtyOnCpu) {
      gpuEmxMemcpyCpuToGpu_creal32_T(&gpu_xdft, xdft);
    }
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_psidft, psidft);
    ec_CWTsingle_kernel13<<<grid, block>>>(gpu_xdft, Nfilt, M, N, k, gpu_psidft,
                                           gpu_cfsdft);
    gpuEmxMemcpyGpuToCpu_real32_T(psidft, &gpu_psidft);
  }
  emxFree_creal32_T(&xdft);
  if ((cfsdft->size[0] == 0) || (cfsdft->size[1] == 0)) {
    for (Nfilt = 0; Nfilt < 2; Nfilt++) {
      xSize[Nfilt] = static_cast<uint32_T>(cfsdft->size[Nfilt]);
      xSize_dirtyOnCpu = true;
    }
    N = cfsdft->size[1];
    if (xSize_dirtyOnCpu) {
      hipMemcpy(*gpu_xSize, xSize, 8UL, hipMemcpyHostToDevice);
    }
    ec_CWTsingle_kernel16<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(N,
                                                                   *gpu_xSize);
    k = cfstmp->size[0] * cfstmp->size[1];
    hipMemcpy(xSize, *gpu_xSize, 8UL, hipMemcpyDeviceToHost);
    cfstmp->size[0] = static_cast<int32_T>(xSize[0]);
    cfstmp->size[1] = cfsdft->size[1];
    emxEnsureCapacity_creal32_T(cfstmp, k, &c_emlrtRTEI);
    gpuEmxEnsureCapacity_creal32_T(cfstmp, &gpu_cfstmp);
    N = static_cast<int32_T>(xSize[0]) * static_cast<int32_T>(xSize[1]) - 1;
    b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                           &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      ec_CWTsingle_kernel17<<<grid, block>>>(N, gpu_cfstmp);
    }
  } else {
    hipfftHandle c_fftPlanHandle;
    N = cfsdft->size[1];
    c_fftPlanHandle = acquireCUFFTPlan(1, &N, &N, cfsdft->size[0], 1, HIPFFT_C2C,
                                       cfsdft->size[0]);
    hipfftExecC2C(c_fftPlanHandle, (hipfftComplex *)&gpu_cfsdft.data[0],
                 (hipfftComplex *)&gpu_cfsdft.data[0], HIPFFT_BACKWARD);
    k = cfstmp->size[0] * cfstmp->size[1];
    cfstmp->size[0] = cfsdft->size[0];
    cfstmp->size[1] = cfsdft->size[1];
    emxEnsureCapacity_creal32_T(cfstmp, k, &c_emlrtRTEI);
    gpuEmxEnsureCapacity_creal32_T(cfstmp, &gpu_cfstmp);
    N = cfsdft->size[1];
    ec_CWTsingle_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        N, b_gpu_cfsdft);
    N = cfsdft->size[0] * cfsdft->size[1] - 1;
    b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                           &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      ec_CWTsingle_kernel15<<<grid, block>>>(b_gpu_cfsdft, gpu_cfsdft, N,
                                             gpu_cfstmp);
    }
  }
  emxFree_creal32_T(&cfsdft);
  if (Npad > 0) {
    N = cfs->size[0];
    Nfilt = cfstmp->size[0];
    b_dirtyOnGpu = mwGetLaunchParameters1D(
        static_cast<real_T>(((Norig - 1) + 1L) * (Ns + 1L)), &grid, &block,
        1024U, 65535U);
    if (b_dirtyOnGpu) {
      ec_CWTsingle_kernel19<<<grid, block>>>(gpu_cfstmp, Npad, Norig, Ns, N,
                                             Nfilt, gpu_cfs);
    }
  } else {
    N = cfs->size[0];
    Nfilt = cfstmp->size[0];
    b_dirtyOnGpu = mwGetLaunchParameters1D(
        static_cast<real_T>(((Norig - 1) + 1L) * (Ns + 1L)), &grid, &block,
        1024U, 65535U);
    if (b_dirtyOnGpu) {
      ec_CWTsingle_kernel18<<<grid, block>>>(gpu_cfstmp, Norig, Ns, N, Nfilt,
                                             gpu_cfs);
    }
  }
  emxFree_creal32_T(&cfstmp);
  if (cf->size[0] == 0) {
    maxwavcf = rtNaNF;
  } else {
    Npad = cf->size[0];
    maxwavcf = cf->data[0];
    for (Nfilt = 0; Nfilt <= Npad - 2; Nfilt++) {
      if (std::isnan(cf->data[Nfilt + 1])) {
        IsOdd = false;
      } else if (std::isnan(maxwavcf)) {
        IsOdd = true;
      } else {
        IsOdd = (maxwavcf < cf->data[Nfilt + 1]);
      }
      if (IsOdd) {
        maxwavcf = cf->data[Nfilt + 1];
      }
    }
  }
  emxInit_real_T(&b_coi, 1, &c_emlrtRTEI, true);
  k = b_coi->size[0];
  b_coi->size[0] = x->size[0];
  emxEnsureCapacity_real_T(b_coi, k, &c_emlrtRTEI);
  gpuEmxEnsureCapacity_real_T(b_coi, &gpu_coi);
  IsOdd = (x->size[0] -
               (static_cast<int32_T>(static_cast<uint32_T>(x->size[0]) >> 1)
                << 1) ==
           1);
  M = static_cast<int32_T>(std::round(static_cast<real_T>(x->size[0]) / 2.0));
  b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>((Norig - 1) + 1L),
                                         &grid, &block, 1024U, 65535U);
  if (b_dirtyOnGpu) {
    ec_CWTsingle_kernel20<<<grid, block>>>(maxwavcf, 1.0 / fs, IsOdd, M, Norig,
                                           gpu_coi);
  }
  k = coi->size[0];
  coi->size[0] = b_coi->size[0];
  emxEnsureCapacity_real32_T(coi, k, &c_emlrtRTEI);
  N = b_coi->size[0] - 1;
  b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                         &block, 1024U, 65535U);
  if (b_dirtyOnGpu) {
    gpuEmxMemcpyCpuToGpu_real32_T(&b_gpu_coi, coi);
    ec_CWTsingle_kernel21<<<grid, block>>>(gpu_coi, N, b_gpu_coi);
    coi_dirtyOnGpu = true;
  }
  emxFree_real_T(&b_coi);
  k = freqs->size[0];
  freqs->size[0] = cf->size[0];
  emxEnsureCapacity_real32_T(freqs, k, &i_emlrtRTEI);
  N = cf->size[0] - 1;
  b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                         &block, 1024U, 65535U);
  if (b_dirtyOnGpu) {
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_freqs, freqs);
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_cf, cf);
    ec_CWTsingle_kernel22<<<grid, block>>>(N, gpu_cf, gpu_freqs);
    freqs_dirtyOnGpu = true;
    gpuEmxMemcpyGpuToCpu_real32_T(cf, &gpu_cf);
  }
  N = cfs->size[0] * cfs->size[1] - 1;
  for (Nfilt = 0; Nfilt < 2; Nfilt++) {
    xSize[Nfilt] = static_cast<uint32_T>(cfs->size[Nfilt]);
  }
  emxInit_real32_T(&d_y, 2, &t_emlrtRTEI, true);
  k = d_y->size[0] * d_y->size[1];
  d_y->size[0] = static_cast<int32_T>(xSize[0]);
  d_y->size[1] = static_cast<int32_T>(xSize[1]);
  emxEnsureCapacity_real32_T(d_y, k, &k_emlrtRTEI);
  gpuEmxEnsureCapacity_real32_T(d_y, &gpu_y);
  b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                         &block, 1024U, 65535U);
  if (b_dirtyOnGpu) {
    ec_CWTsingle_kernel23<<<grid, block>>>(gpu_cfs, N, gpu_y);
  }
  emxFree_creal32_T(&cfs);
  k = wt->size[0] * wt->size[1];
  wt->size[0] = d_y->size[1];
  wt->size[1] = d_y->size[0];
  emxEnsureCapacity_real32_T(wt, k, &l_emlrtRTEI);
  N = d_y->size[0] - 1;
  Nfilt = d_y->size[1] - 1;
  k = wt->size[0];
  M = d_y->size[0];
  b_dirtyOnGpu =
      mwGetLaunchParameters1D(static_cast<real_T>((Nfilt + 1L) * (N + 1L)),
                              &grid, &block, 1024U, 65535U);
  if (b_dirtyOnGpu) {
    gpuEmxMemcpyCpuToGpu_real32_T(&gpu_wt, wt);
    ec_CWTsingle_kernel24<<<grid, block>>>(gpu_y, Nfilt, N, k, M, gpu_wt);
    wt_dirtyOnCpu = false;
    wt_dirtyOnGpu = true;
  }
  emxFree_real32_T(&d_y);
  if (doAvg) {
    emxInit_real32_T(&e_y, 1, &u_emlrtRTEI, true);
    Npad = wt->size[1];
    if (wt->size[1] == 0) {
      for (Nfilt = 0; Nfilt < 2; Nfilt++) {
        xSize[Nfilt] = static_cast<uint32_T>(wt->size[Nfilt]);
      }
      k = e_y->size[0];
      e_y->size[0] = static_cast<int32_T>(xSize[0]);
      emxEnsureCapacity_real32_T(e_y, k, &n_emlrtRTEI);
      gpuEmxEnsureCapacity_real32_T(e_y, &b_gpu_y);
      b_dirtyOnGpu = mwGetLaunchParameters1D(
          static_cast<real_T>((static_cast<int32_T>(xSize[0]) - 1) + 1L), &grid,
          &block, 1024U, 65535U);
      if (b_dirtyOnGpu) {
        ec_CWTsingle_kernel27<<<grid, block>>>(
            static_cast<int32_T>(xSize[0]) - 1, b_gpu_y);
      }
    } else {
      M = wt->size[0] - 1;
      k = e_y->size[0];
      e_y->size[0] = wt->size[0];
      emxEnsureCapacity_real32_T(e_y, k, &m_emlrtRTEI);
      gpuEmxEnsureCapacity_real32_T(e_y, &b_gpu_y);
      b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(M + 1L), &grid,
                                             &block, 1024U, 65535U);
      if (b_dirtyOnGpu) {
        if (wt_dirtyOnCpu) {
          gpuEmxMemcpyCpuToGpu_real32_T(&gpu_wt, wt);
        }
        wt_dirtyOnCpu = false;
        ec_CWTsingle_kernel25<<<grid, block>>>(gpu_wt, M, b_gpu_y);
      }
      for (k = 0; k <= Npad - 2; k++) {
        b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(M + 1L),
                                               &grid, &block, 1024U, 65535U);
        if (b_dirtyOnGpu) {
          if (wt_dirtyOnCpu) {
            gpuEmxMemcpyCpuToGpu_real32_T(&gpu_wt, wt);
          }
          wt_dirtyOnCpu = false;
          ec_CWTsingle_kernel26<<<grid, block>>>(gpu_wt, (k + 1) * (M + 1), M,
                                                 b_gpu_y);
        }
      }
    }
    maxwavcf = static_cast<real32_T>(wt->size[1]);
    emxInit_real32_T(&b_wt, 1, &q_emlrtRTEI, true);
    k = b_wt->size[0];
    b_wt->size[0] = e_y->size[0];
    emxEnsureCapacity_real32_T(b_wt, k, &o_emlrtRTEI);
    gpuEmxEnsureCapacity_real32_T(b_wt, &b_gpu_wt);
    N = e_y->size[0] - 1;
    b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                           &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      ec_CWTsingle_kernel28<<<grid, block>>>(maxwavcf, b_gpu_y, N, b_gpu_wt);
    }
    emxFree_real32_T(&e_y);
    k = wt->size[0] * wt->size[1];
    wt->size[0] = b_wt->size[0];
    wt->size[1] = 1;
    emxEnsureCapacity_real32_T(wt, k, &p_emlrtRTEI);
    if (!wt_dirtyOnCpu) {
      gpuEmxEnsureCapacity_real32_T(wt, &gpu_wt);
    }
    N = b_wt->size[0] - 1;
    b_dirtyOnGpu = mwGetLaunchParameters1D(static_cast<real_T>(N + 1L), &grid,
                                           &block, 1024U, 65535U);
    if (b_dirtyOnGpu) {
      if (wt_dirtyOnCpu) {
        gpuEmxMemcpyCpuToGpu_real32_T(&gpu_wt, wt);
      }
      ec_CWTsingle_kernel29<<<grid, block>>>(b_gpu_wt, N, gpu_wt);
      wt_dirtyOnGpu = true;
    }
    emxFree_real32_T(&b_wt);
  }
  //  % Get number of freqs
  //  nFrames = size(x,1);
  //  freqs = cwtfilterbank(SignalLength=nFrames,SamplingFrequency=fs,...
  //      VoicesPerOctave=freqsPerOctave,FrequencyLimits=freqLims);
  //  freqs = freqs.centerFrequencies';
  //
  //  % Initialize
  //  coder.gpu.kernelfun; % Add kernelfun pragma to trigger kernel creation
  //  nChs = size(x,2);
  //  nFreqs = numel(freqs);
  //  if ~doAvg
  //      wt = coder.nullcopy(zeros(nFrames,nChs,nFreqs));
  //  else
  //      wt = coder.nullcopy(zeros(nFrames,nChs));
  //  end
  //
  //  % Do CWT
  //  coder.gpu.kernel();
  //  if ~doAvg
  //      for ch = 1:nChs
  //          wt(:,ch,:) = abs(cwt(x(:,ch),'morse',fs,...
  //              VoicesPerOctave=freqsPerOctave,FrequencyLimits=freqLims))';
  //      end
  //  else
  //      for ch = 1:nChs
  //          wt(:,ch) = mean(abs(cwt(x(:,ch),'morse',fs,...
  //              VoicesPerOctave=freqsPerOctave,FrequencyLimits=freqLims)),1)';
  //      end
  //  end
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  if (wt_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real32_T(wt, &gpu_wt);
  }
  if (freqs_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real32_T(freqs, &gpu_freqs);
  }
  if (coi_dirtyOnGpu) {
    gpuEmxMemcpyGpuToCpu_real32_T(coi, &b_gpu_coi);
  }
  mwCudaFree(&(*gpu_freqLims)[0]);
  mwCudaFree(&(*gpu_b)[0]);
  gpuEmxFree_real32_T(&gpu_x);
  gpuEmxFree_real32_T(&gpu_xv);
  gpuEmxFree_real_T(&gpu_omega);
  gpuEmxFree_creal32_T(&gpu_xdft);
  gpuEmxFree_real32_T(&gpu_psidft);
  gpuEmxFree_creal32_T(&gpu_cfsdft);
  mwCudaFree(&(*gpu_xSize)[0]);
  mwCudaFree(b_gpu_cfsdft);
  gpuEmxFree_creal32_T(&gpu_cfstmp);
  gpuEmxFree_real_T(&gpu_scales);
  gpuEmxFree_creal32_T(&gpu_cfs);
  gpuEmxFree_real32_T(&gpu_cf);
  gpuEmxFree_real_T(&gpu_coi);
  gpuEmxFree_real32_T(&b_gpu_coi);
  gpuEmxFree_real32_T(&gpu_freqs);
  gpuEmxFree_real32_T(&gpu_y);
  gpuEmxFree_real32_T(&gpu_wt);
  gpuEmxFree_real32_T(&b_gpu_y);
  gpuEmxFree_real32_T(&b_gpu_wt);
}

// End of code generation (ec_CWTsingle.cu)

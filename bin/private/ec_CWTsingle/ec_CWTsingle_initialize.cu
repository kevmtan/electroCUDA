//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_CWTsingle_initialize.cu
//
// Code generation for function 'ec_CWTsingle_initialize'
//

// Include files
#include "ec_CWTsingle_initialize.h"
#include "_coder_ec_CWTsingle_mex.h"
#include "cwt.h"
#include "ec_CWTsingle_data.h"
#include "rt_nonfinite.h"
#include "MWMemoryManager.hpp"

// Function Declarations
static void ec_CWTsingle_once();

// Function Definitions
static void ec_CWTsingle_once()
{
  mex_InitInfAndNan();
  mwMemoryManagerInit(256U, 1U, 8U, 2048U);
  cwt_init();
}

void ec_CWTsingle_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "wavelet_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    ec_CWTsingle_once();
  }
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (ec_CWTsingle_initialize.cu)

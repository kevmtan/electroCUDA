//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_CWTsingle_terminate.cu
//
// Code generation for function 'ec_CWTsingle_terminate'
//

// Include files
#include "ec_CWTsingle_terminate.h"
#include "_coder_ec_CWTsingle_mex.h"
#include "cwt.h"
#include "ec_CWTsingle_data.h"
#include "rt_nonfinite.h"
#include "MWMemoryManager.hpp"

// Function Definitions
void ec_CWTsingle_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  cwt_free();
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void ec_CWTsingle_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  mwMemoryManagerTerminate();
}

// End of code generation (ec_CWTsingle_terminate.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_CWTdouble_initialize.cu
//
// Code generation for function 'ec_CWTdouble_initialize'
//

// Include files
#include "ec_CWTdouble_initialize.h"
#include "_coder_ec_CWTdouble_mex.h"
#include "cwt.h"
#include "ec_CWTdouble_data.h"
#include "rt_nonfinite.h"
#include "MWMemoryManager.hpp"

// Function Declarations
static void ec_CWTdouble_once();

// Function Definitions
static void ec_CWTdouble_once()
{
  mex_InitInfAndNan();
  mwMemoryManagerInit(256U, 1U, 8U, 2048U);
  cwt_init();
}

void ec_CWTdouble_initialize()
{
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "wavelet_toolbox", 2);
  if (emlrtFirstTimeR2012b(emlrtRootTLSGlobal)) {
    ec_CWTdouble_once();
  }
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (ec_CWTdouble_initialize.cu)

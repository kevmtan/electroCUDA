//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// ec_CWTdouble_terminate.cu
//
// Code generation for function 'ec_CWTdouble_terminate'
//

// Include files
#include "ec_CWTdouble_terminate.h"
#include "_coder_ec_CWTdouble_mex.h"
#include "cwt.h"
#include "ec_CWTdouble_data.h"
#include "rt_nonfinite.h"
#include "MWMemoryManager.hpp"

// Function Definitions
void ec_CWTdouble_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  cwt_free();
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void ec_CWTdouble_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  mwMemoryManagerTerminate();
}

// End of code generation (ec_CWTdouble_terminate.cu)
